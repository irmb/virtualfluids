#include "hip/hip_runtime.h"
#include "CumulantChimeraPreCompiled.h"

#include <cmath>

#include <basics/Core/DataTypes.h>
#include <basics/Core/RealConstants.h>

#include "D3Q27.h"
#include "Chimera.h"
#include "MacroscopicQuantities.h"

namespace VF
{
namespace LBM
{


//////////////////////////////////////////////////////////////////////////
//! Cumulant K17 Kernel is based on \ref
//! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
//! and \ref
//! <a href="https://doi.org/10.1016/j.jcp.2017.07.004"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.07.004 ]</b></a>
//////////////////////////////////////////////////////////////////////////
__host__ __device__ void cumulantChimera(Distribution27& distribution, real omega, real* forces)
{
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Read distributions: style of reading and writing the distributions from/to 
    //! stored arrays dependent on timestep is based on the esoteric twist algorithm
    //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
    //!
    real mfcbb = distribution.f[DIR::PZZ];
    real mfabb = distribution.f[DIR::MZZ];
    real mfbcb = distribution.f[DIR::ZPZ];
    real mfbab = distribution.f[DIR::ZMZ];
    real mfbbc = distribution.f[DIR::ZZP];
    real mfbba = distribution.f[DIR::ZZM];
    real mfccb = distribution.f[DIR::PPZ];
    real mfaab = distribution.f[DIR::MMZ];
    real mfcab = distribution.f[DIR::PMZ];
    real mfacb = distribution.f[DIR::MPZ];
    real mfcbc = distribution.f[DIR::PZP];
    real mfaba = distribution.f[DIR::MZM];
    real mfcba = distribution.f[DIR::PZM];
    real mfabc = distribution.f[DIR::MZP];
    real mfbcc = distribution.f[DIR::ZPP];
    real mfbaa = distribution.f[DIR::ZMM];
    real mfbca = distribution.f[DIR::ZPM];
    real mfbac = distribution.f[DIR::ZMP];
    real mfccc = distribution.f[DIR::PPP];
    real mfacc = distribution.f[DIR::MPP];
    real mfcac = distribution.f[DIR::PMP];
    real mfaac = distribution.f[DIR::MMP];
    real mfcca = distribution.f[DIR::PPM];
    real mfaca = distribution.f[DIR::MPM];
    real mfcaa = distribution.f[DIR::PMM];
    real mfaaa = distribution.f[DIR::MMM];
    real mfbbb = distribution.f[DIR::ZZZ];

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Calculate density and velocity using pyramid summation for low round-off errors as in Eq. (J1)-(J3) \ref
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
    //!
    real drho =
        ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
        (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
        ((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb; 
    real rho = c1o1 + drho;
    real OOrho = c1o1 / rho;    
    real vvx = 
        ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
        (((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
        (mfcbb - mfabb)) * OOrho;
    real vvy = 
        ((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
        (((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
        (mfbcb - mfbab)) * OOrho;
    real vvz = 
        ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
        (((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
        (mfbbc - mfbba)) * OOrho;
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Add half of the acceleration (body force) to the velocity as in Eq. (42) \ref
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
    //!
    vvx += forces[0] * c1o2;
    vvy += forces[1] * c1o2;
    vvz += forces[2] * c1o2;
    ////////////////////////////////////////////////////////////////////////////////////
    // calculate the square of velocities for this lattice node
    real vx2 = vvx*vvx;
    real vy2 = vvy*vvy;
    real vz2 = vvz*vvz;
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Set relaxation limiters for third order cumulants to default value \f$ \lambda=0.001 \f$ according to section 6 in \ref
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //!
    real wadjust;
    real qudricLimitP = c1o100;
    real qudricLimitM = c1o100;
    real qudricLimitD = c1o100;
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Chimera transform from well conditioned distributions to central moments as defined in Appendix J in \ref
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
    //! see also Eq. (6)-(14) in \ref
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //!
    ////////////////////////////////////////////////////////////////////////////////////
    // Z - Dir
    VF::LBM::forwardInverseChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, c36o1, c1o36);
    VF::LBM::forwardInverseChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2,  c9o1,  c1o9);
    VF::LBM::forwardInverseChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c36o1, c1o36);
    VF::LBM::forwardInverseChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2,  c9o1,  c1o9);
    VF::LBM::forwardInverseChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2,  c9o4,  c4o9);
    VF::LBM::forwardInverseChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2,  c9o1,  c1o9);
    VF::LBM::forwardInverseChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c36o1, c1o36);
    VF::LBM::forwardInverseChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2,  c9o1,  c1o9);
    VF::LBM::forwardInverseChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c36o1, c1o36);   
    ////////////////////////////////////////////////////////////////////////////////////
    // Y - Dir
    VF::LBM::forwardInverseChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2,  c6o1,  c1o6);
    VF::LBM::forwardChimera(            mfaab, mfabb, mfacb, vvy, vy2);
    VF::LBM::forwardInverseChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c18o1, c1o18);
    VF::LBM::forwardInverseChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2,  c3o2,  c2o3);
    VF::LBM::forwardChimera(            mfbab, mfbbb, mfbcb, vvy, vy2);
    VF::LBM::forwardInverseChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2,  c9o2,  c2o9);
    VF::LBM::forwardInverseChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2,  c6o1,  c1o6);
    VF::LBM::forwardChimera(            mfcab, mfcbb, mfccb, vvy, vy2);
    VF::LBM::forwardInverseChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c18o1, c1o18);   
    ////////////////////////////////////////////////////////////////////////////////////
    // X - Dir
    VF::LBM::forwardInverseChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o1, c1o1);
    VF::LBM::forwardChimera(            mfaba, mfbba, mfcba, vvx, vx2);
    VF::LBM::forwardInverseChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c3o1, c1o3);
    VF::LBM::forwardChimera(            mfaab, mfbab, mfcab, vvx, vx2);
    VF::LBM::forwardChimera(            mfabb, mfbbb, mfcbb, vvx, vx2);
    VF::LBM::forwardChimera(            mfacb, mfbcb, mfccb, vvx, vx2);
    VF::LBM::forwardInverseChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c3o1, c1o3);
    VF::LBM::forwardChimera(            mfabc, mfbbc, mfcbc, vvx, vx2);
    VF::LBM::forwardInverseChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c3o1, c1o9); 
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Setting relaxation rates for non-hydrodynamic cumulants (default values). Variable names and equations    according to
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //!  => [NAME IN PAPER]=[NAME IN CODE]=[DEFAULT VALUE].
    //!  - Trace of second order cumulants \f$ C_{200}+C_{020}+C_{002} \f$ used to adjust bulk  viscosity:\f$\omega_2=OxxPyyPzz=1.0 \f$.
    //!  - Third order cumulants \f$ C_{120}+C_{102}, C_{210}+C_{012}, C_{201}+C_{021} \f$: \f$ \omega_3=OxyyPxzz   \f$ set according to Eq. (111) with simplifications assuming \f$ \omega_2=1.0\f$.
    //!  - Third order cumulants \f$ C_{120}-C_{102}, C_{210}-C_{012}, C_{201}-C_{021} \f$: \f$ \omega_4 =  OxyyMxzz \f$ set according to Eq. (112) with simplifications assuming \f$ \omega_2 = 1.0\f$.
    //!  - Third order cumulants \f$ C_{111} \f$: \f$ \omega_5 = Oxyz \f$ set according to Eq. (113) with   simplifications assuming \f$ \omega_2 = 1.0\f$  (modify for different bulk viscosity).
    //!  - Fourth order cumulants \f$ C_{220}, C_{202}, C_{022}, C_{211}, C_{121}, C_{112} \f$: for simplification  all set to the same default value \f$ \omega_6=\omega_7=\omega_8=O4=1.0 \f$.
    //!  - Fifth order cumulants \f$ C_{221}, C_{212}, C_{122}\f$: \f$\omega_9=O5=1.0\f$.
    //!  - Sixth order cumulant \f$ C_{222}\f$: \f$\omega_{10}=O6=1.0\f$.
    //!
    ////////////////////////////////////////////////////////////
    //2.
    real OxxPyyPzz = c1o1;
    ////////////////////////////////////////////////////////////
    //3.
    real OxyyPxzz = c8o1  * (-c2o1 + omega) * ( c1o1 + c2o1*omega) / (-c8o1 - c14o1*omega + c7o1*omega*omega);
    real OxyyMxzz = c8o1  * (-c2o1 + omega) * (-c7o1 + c4o1*omega) / (c56o1 - c50o1*omega + c9o1*omega*omega);
    real Oxyz     = c24o1 * (-c2o1 + omega) * (-c2o1 - c7o1*omega + c3o1*omega*omega) / (c48o1 + c152o1*omega - c130o1*omega*omega + c29o1*omega*omega*omega);
    ////////////////////////////////////////////////////////////
    //4.
    real O4 = c1o1;
    ////////////////////////////////////////////////////////////
    //5.
    real O5 = c1o1;
    ////////////////////////////////////////////////////////////
    //6.
    real O6 = c1o1; 
    ////////////////////////////////////////////////////////////////////////////////////
    //! - A and B: parameters for fourth order convergence of the diffusion term according to Eq. (114) and (115) 
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //! with simplifications assuming \f$ \omega_2 = 1.0 \f$ (modify for different bulk viscosity).
    //!
    real A = (c4o1 + c2o1*omega - c3o1*omega*omega) / (c2o1 - c7o1*omega + c5o1*omega*omega);
    real B = (c4o1 + c28o1*omega - c14o1*omega*omega) / (c6o1 - c21o1*omega + c15o1*omega*omega);   
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Compute cumulants from central moments according to Eq. (20)-(23) in
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //!
    ////////////////////////////////////////////////////////////
    //4.
    real CUMcbb = mfcbb - ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) * OOrho;
    real CUMbcb = mfbcb - ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) * OOrho;
    real CUMbbc = mfbbc - ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) * OOrho;  
    real CUMcca = mfcca - (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) * OOrho - c1o9*(drho   * OOrho));
    real CUMcac = mfcac - (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) * OOrho - c1o9*(drho   * OOrho));
    real CUMacc = mfacc - (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) * OOrho - c1o9*(drho   * OOrho));
    ////////////////////////////////////////////////////////////
    //5.
    real CUMbcc = mfbcc - ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba *  mfabc)) + c1o3 * (mfbca + mfbac)) * OOrho;
    real CUMcbc = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba *  mfbac)) + c1o3 * (mfcba + mfabc)) * OOrho;
    real CUMccb = mfccb - ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb *  mfcba)) + c1o3 * (mfacb + mfcab)) * OOrho;
    ////////////////////////////////////////////////////////////
    //6.
    real CUMccc = mfccc + ((-c4o1 *  mfbbb * mfbbb
        - (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
        - c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
        - c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) * OOrho
        + (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
        + c2o1 * (mfcaa * mfaca * mfaac)
        + c16o1 *  mfbba * mfbab * mfabb) * OOrho * OOrho
        - c1o3 * (mfacc + mfcac + mfcca) * OOrho
        - c1o9 * (mfcaa + mfaca + mfaac) * OOrho
        + (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
        + (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) * OOrho * OOrho  * c2o3
        + c1o27*((drho * drho - drho) * OOrho * OOrho));    
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Compute linear combinations of second and third order cumulants
    //!
    ////////////////////////////////////////////////////////////
    //2.
    real mxxPyyPzz = mfcaa + mfaca + mfaac;
    real mxxMyy = mfcaa - mfaca;
    real mxxMzz = mfcaa - mfaac;
    ////////////////////////////////////////////////////////////
    //3.
    real mxxyPyzz = mfcba + mfabc;
    real mxxyMyzz = mfcba - mfabc;  
    real mxxzPyyz = mfcab + mfacb;
    real mxxzMyyz = mfcab - mfacb;  
    real mxyyPxzz = mfbca + mfbac;
    real mxyyMxzz = mfbca - mfbac;  
    ////////////////////////////////////////////////////////////////////////////////////
    //incl. correction
    ////////////////////////////////////////////////////////////
    //! - Compute velocity  gradients from second order cumulants according to Eq. (27)-(32)
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //! Further explanations of the correction in viscosity in Appendix H of
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
    //! Note that the division by rho is omitted here as we need rho times the gradients later.
    //!
    real Dxy = -c3o1*omega*mfbba;
    real Dxz = -c3o1*omega*mfbab;
    real Dyz = -c3o1*omega*mfabb;
    real dxux = c1o2 * (-omega) *(mxxMyy + mxxMzz) + c1o2 *  OxxPyyPzz * (mfaaa - mxxPyyPzz);
    real dyuy = dxux + omega * c3o2 * mxxMyy;
    real dzuz = dxux + omega * c3o2 * mxxMzz;
    ////////////////////////////////////////////////////////////
    //! - Relaxation of second order cumulants with correction terms according to Eq. (33)-(35) in
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //!
    mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz) - c3o1 * (c1o1 - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2  * dzuz);
    mxxMyy    += omega * (-mxxMyy) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy);
    mxxMzz    += omega * (-mxxMzz) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz);   
    ////////////////////////////////////////////////////////////////////////////////////
    ////no correction
    //mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz);
    //mxxMyy += -(-omega) * (-mxxMyy);
    //mxxMzz += -(-omega) * (-mxxMzz);
    //////////////////////////////////////////////////////////////////////////
    mfabb += omega * (-mfabb);
    mfbab += omega * (-mfbab);
    mfbba += omega * (-mfbba);  
    ////////////////////////////////////////////////////////////////////////////////////
    //relax
    //////////////////////////////////////////////////////////////////////////
    // incl. limiter
    //! - Relaxation of third order cumulants including limiter according to Eq. (116)-(123)
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //!
    wadjust   = Oxyz + (c1o1 - Oxyz)*abs_internal(mfbbb) / (abs_internal(mfbbb) + qudricLimitD);
    mfbbb    += wadjust * (-mfbbb);
    wadjust   = OxyyPxzz + (c1o1 - OxyyPxzz)*abs_internal(mxxyPyzz) / (abs_internal(mxxyPyzz) + qudricLimitP);
    mxxyPyzz += wadjust * (-mxxyPyzz);
    wadjust   = OxyyMxzz + (c1o1 - OxyyMxzz)*abs_internal(mxxyMyzz) / (abs_internal(mxxyMyzz) + qudricLimitM);
    mxxyMyzz += wadjust * (-mxxyMyzz);
    wadjust   = OxyyPxzz + (c1o1 - OxyyPxzz)*abs_internal(mxxzPyyz) / (abs_internal(mxxzPyyz) + qudricLimitP);
    mxxzPyyz += wadjust * (-mxxzPyyz);
    wadjust   = OxyyMxzz + (c1o1 - OxyyMxzz)*abs_internal(mxxzMyyz) / (abs_internal(mxxzMyyz) + qudricLimitM);
    mxxzMyyz += wadjust * (-mxxzMyyz);
    wadjust   = OxyyPxzz + (c1o1 - OxyyPxzz)*abs_internal(mxyyPxzz) / (abs_internal(mxyyPxzz) + qudricLimitP);
    mxyyPxzz += wadjust * (-mxyyPxzz);
    wadjust   = OxyyMxzz + (c1o1 - OxyyMxzz)*abs_internal(mxyyMxzz) / (abs_internal(mxyyMxzz) + qudricLimitM);
    mxyyMxzz += wadjust * (-mxyyMxzz);
    //////////////////////////////////////////////////////////////////////////
    // no limiter
    //mfbbb += OxyyMxzz * (-mfbbb);
    //mxxyPyzz += OxyyPxzz * (-mxxyPyzz);
    //mxxyMyzz += OxyyMxzz * (-mxxyMyzz);
    //mxxzPyyz += OxyyPxzz * (-mxxzPyyz);
    //mxxzMyyz += OxyyMxzz * (-mxxzMyyz);
    //mxyyPxzz += OxyyPxzz * (-mxyyPxzz);
    //mxyyMxzz += OxyyMxzz * (-mxyyMxzz);   
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Compute inverse linear combinations of second and third order cumulants
    //!
    mfcaa = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
    mfaca = c1o3 * (-c2o1*  mxxMyy + mxxMzz + mxxPyyPzz);
    mfaac = c1o3 * (mxxMyy - c2o1* mxxMzz + mxxPyyPzz); 
    mfcba = ( mxxyMyzz + mxxyPyzz) * c1o2;
    mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
    mfcab = ( mxxzMyyz + mxxzPyyz) * c1o2;
    mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
    mfbca = ( mxyyMxzz + mxyyPxzz) * c1o2;
    mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;
    //////////////////////////////////////////////////////////////////////////  
    //////////////////////////////////////////////////////////////////////////
    //4.
    // no limiter
    //! - Relax fourth order cumulants to modified equilibrium for fourth order convergence of diffusion according  to Eq. (43)-(48)
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //!
    CUMacc = -O4*(c1o1 / omega - c1o2) * (dyuy + dzuz) * c2o3 * A + (c1o1 - O4) * (CUMacc);
    CUMcac = -O4*(c1o1 / omega - c1o2) * (dxux + dzuz) * c2o3 * A + (c1o1 - O4) * (CUMcac);
    CUMcca = -O4*(c1o1 / omega - c1o2) * (dyuy + dxux) * c2o3 * A + (c1o1 - O4) * (CUMcca);
    CUMbbc = -O4*(c1o1 / omega - c1o2) * Dxy           * c1o3 * B + (c1o1 - O4) * (CUMbbc);
    CUMbcb = -O4*(c1o1 / omega - c1o2) * Dxz           * c1o3 * B + (c1o1 - O4) * (CUMbcb);
    CUMcbb = -O4*(c1o1 / omega - c1o2) * Dyz           * c1o3 * B + (c1o1 - O4) * (CUMcbb); 
    //////////////////////////////////////////////////////////////////////////
    //5.
    CUMbcc += O5 * (-CUMbcc);
    CUMcbc += O5 * (-CUMcbc);
    CUMccb += O5 * (-CUMccb);   
    //////////////////////////////////////////////////////////////////////////
    //6.
    CUMccc += O6 * (-CUMccc);   
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Compute central moments from post collision cumulants according to Eq. (53)-(56) in
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //! 
    //////////////////////////////////////////////////////////////////////////
    //4.
    mfcbb = CUMcbb + c1o3*((c3o1*mfcaa + c1o1) * mfabb + c6o1 * mfbba * mfbab) * OOrho;
    mfbcb = CUMbcb + c1o3*((c3o1*mfaca + c1o1) * mfbab + c6o1 * mfbba * mfabb) * OOrho;
    mfbbc = CUMbbc + c1o3*((c3o1*mfaac + c1o1) * mfbba + c6o1 * mfbab * mfabb) * OOrho; 
    mfcca = CUMcca + (((mfcaa * mfaca + c2o1 * mfbba * mfbba)*c9o1 + c3o1 * (mfcaa + mfaca)) * OOrho - (drho *  OOrho))*c1o9;
    mfcac = CUMcac + (((mfcaa * mfaac + c2o1 * mfbab * mfbab)*c9o1 + c3o1 * (mfcaa + mfaac)) * OOrho - (drho *  OOrho))*c1o9;
    mfacc = CUMacc + (((mfaac * mfaca + c2o1 * mfabb * mfabb)*c9o1 + c3o1 * (mfaac + mfaca)) * OOrho - (drho *  OOrho))*c1o9; 
    //////////////////////////////////////////////////////////////////////////
    //5.
    mfbcc = CUMbcc + c1o3 *(c3o1*(mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb +    mfbba * mfabc)) + (mfbca + mfbac)) * OOrho;
    mfcbc = CUMcbc + c1o3 *(c3o1*(mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab +    mfbba * mfbac)) + (mfcba + mfabc)) * OOrho;
    mfccb = CUMccb + c1o3 *(c3o1*(mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca +    mfabb * mfcba)) + (mfacb + mfcab)) * OOrho; 
    //////////////////////////////////////////////////////////////////////////
    //6.
    mfccc =	CUMccc - ((-c4o1 *  mfbbb * mfbbb
            - (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
            - c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
            - c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) * OOrho
            + (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
                + c2o1 * (mfcaa * mfaca * mfaac)
                + c16o1 *  mfbba * mfbab * mfabb) * OOrho * OOrho
            - c1o3 * (mfacc + mfcac + mfcca) * OOrho
            - c1o9 * (mfcaa + mfaca + mfaac) * OOrho
            + (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
                + (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) * OOrho * OOrho * c2o3
            + c1o27*((drho * drho - drho) * OOrho * OOrho));    
    ////////////////////////////////////////////////////////////////////////////////////
    //! -  Add acceleration (body force) to first order cumulants according to Eq. (85)-(87) in
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
    //!
    mfbaa = -mfbaa;
    mfaba = -mfaba;
    mfaab = -mfaab; 
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Chimera transform from central moments to well conditioned distributions as defined in Appendix J in
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
    //! see also Eq. (88)-(96) in
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
    //!
    ////////////////////////////////////////////////////////////////////////////////////
    // X - Dir
    VF::LBM::backwardInverseChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o1, c1o1);
    VF::LBM::backwardChimera(            mfaba, mfbba, mfcba, vvx, vx2);
    VF::LBM::backwardInverseChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c3o1, c1o3);
    VF::LBM::backwardChimera(            mfaab, mfbab, mfcab, vvx, vx2);
    VF::LBM::backwardChimera(            mfabb, mfbbb, mfcbb, vvx, vx2);
    VF::LBM::backwardChimera(            mfacb, mfbcb, mfccb, vvx, vx2);
    VF::LBM::backwardInverseChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c3o1, c1o3);
    VF::LBM::backwardChimera(            mfabc, mfbbc, mfcbc, vvx, vx2);
    VF::LBM::backwardInverseChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c9o1, c1o9);    
    ////////////////////////////////////////////////////////////////////////////////////
    // Y - Dir
    VF::LBM::backwardInverseChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2,  c6o1,  c1o6);
    VF::LBM::backwardChimera(            mfaab, mfabb, mfacb, vvy, vy2);
    VF::LBM::backwardInverseChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c18o1, c1o18);
    VF::LBM::backwardInverseChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2,  c3o2,  c2o3);
    VF::LBM::backwardChimera(            mfbab, mfbbb, mfbcb, vvy, vy2);
    VF::LBM::backwardInverseChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2,  c9o2,  c2o9);
    VF::LBM::backwardInverseChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2,  c6o1,  c1o6);
    VF::LBM::backwardChimera(            mfcab, mfcbb, mfccb, vvy, vy2);
    VF::LBM::backwardInverseChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c18o1, c1o18);  
    ////////////////////////////////////////////////////////////////////////////////////
    // Z - Dir
    VF::LBM::backwardInverseChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, c36o1, c1o36);
    VF::LBM::backwardInverseChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2,  c9o1,  c1o9);
    VF::LBM::backwardInverseChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c36o1, c1o36);
    VF::LBM::backwardInverseChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2,  c9o1,  c1o9);
    VF::LBM::backwardInverseChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2,  c9o4,  c4o9);
    VF::LBM::backwardInverseChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2,  c9o1,  c1o9);
    VF::LBM::backwardInverseChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c36o1, c1o36);
    VF::LBM::backwardInverseChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2,  c9o1,  c1o9);
    VF::LBM::backwardInverseChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c36o1, c1o36);


    ////////////////////////////////////////////////////////////////////////////////////
    //! - Write distributions: style of reading and writing the distributions from/to 
    //! stored arrays dependent on timestep is based on the esoteric twist algorithm
    //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
    //!
    distribution.f[VF::LBM::DIR::MZZ] = mfcbb;
    distribution.f[VF::LBM::DIR::PZZ] = mfabb;
    distribution.f[VF::LBM::DIR::ZMZ] = mfbcb;
    distribution.f[VF::LBM::DIR::ZPZ] = mfbab;
    distribution.f[VF::LBM::DIR::ZZM] = mfbbc;
    distribution.f[VF::LBM::DIR::ZZP] = mfbba;
    distribution.f[VF::LBM::DIR::MMZ] = mfccb;
    distribution.f[VF::LBM::DIR::PPZ] = mfaab;
    distribution.f[VF::LBM::DIR::MPZ] = mfcab;
    distribution.f[VF::LBM::DIR::PMZ] = mfacb;
    distribution.f[VF::LBM::DIR::MZM] = mfcbc;
    distribution.f[VF::LBM::DIR::PZP] = mfaba;
    distribution.f[VF::LBM::DIR::MZP] = mfcba;
    distribution.f[VF::LBM::DIR::PZM] = mfabc;
    distribution.f[VF::LBM::DIR::ZMM] = mfbcc;
    distribution.f[VF::LBM::DIR::ZPP] = mfbaa;
    distribution.f[VF::LBM::DIR::ZMP] = mfbca;
    distribution.f[VF::LBM::DIR::ZPM] = mfbac;
    distribution.f[VF::LBM::DIR::MMM] = mfccc;
    distribution.f[VF::LBM::DIR::PMM] = mfacc;
    distribution.f[VF::LBM::DIR::MPM] = mfcac;
    distribution.f[VF::LBM::DIR::PPM] = mfaac;
    distribution.f[VF::LBM::DIR::MMP] = mfcca;
    distribution.f[VF::LBM::DIR::PMP] = mfaca;
    distribution.f[VF::LBM::DIR::MPP] = mfcaa;
    distribution.f[VF::LBM::DIR::PPP] = mfaaa;
    distribution.f[VF::LBM::DIR::ZZZ] = mfbbb;
}


}
}

