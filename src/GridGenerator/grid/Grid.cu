#include "hip/hip_runtime.h"
#include "Grid.cuh"

#include "GridGenerator/global.h"
#include <stdio.h>
#include <time.h>

#include <sstream>


#include <GridGenerator/utilities/math/CudaMath.cuh>
#include "distributions/Distribution.h"

#include <GridGenerator/geometries/Vertex/Vertex.cuh>
#include <GridGenerator/geometries/Triangle/Triangle.cuh>
#include <GridGenerator/geometries/BoundingBox/BoundingBox.cuh>

#include <GridGenerator/grid/NodeValues.h>
#include <GridGenerator/grid/distributions/Distribution.h>

#include <GridGenerator/grid/GridStrategy/GridStrategy.h>
#include <utilities/logger/Logger.h>
#include "GridInterface.cuh"


CONSTANT int DIRECTIONS[DIR_END_MAX][DIMENSION];

HOST Grid::Grid(real startX, real startY, real startZ, real endX, real endY, real endZ, real delta, std::shared_ptr<GridStrategy> gridStrategy, Distribution distribution)
    : startX(startX), startY(startY), startZ(startZ), endX(endX), endY(endY), endZ(endZ), delta(delta), field(nullptr), distribution(distribution),
    gridInterface(nullptr), neighborIndexX(nullptr), neighborIndexY(nullptr), neighborIndexZ(nullptr), matrixIndex(nullptr), gridStrategy(gridStrategy)
{
    const real length = endX - startX;
    const real width = endY - startY;
    const real height = endZ - startZ;

    nx = int((length + delta) / delta) + 1; // +1 stopper node
    ny = int((width + delta) / delta) + 1; // +1 stopper node
    nz = int((height + delta) / delta) + 1; // +1 stopper node

    this->size = nx * ny * nz;
    this->reducedSize = size;
    distribution.setSize(size);
}

HOST SPtr<Grid> Grid::makeShared(real startX, real startY, real startZ, real endX, real endY, real endZ, real delta,
                                 std::shared_ptr<GridStrategy> gridStrategy, Distribution d)
{
    SPtr<Grid> grid(new Grid(startX, startY, startZ, endX, endY, endZ, delta, gridStrategy, d));

    gridStrategy->allocateGridMemory(grid);

    *logging::out << logging::Logger::LOW << "-------------------------------------------\n";
    *logging::out << logging::Logger::LOW << "Initial field with fluid. \n";
    *logging::out << logging::Logger::LOW << "-------------------------------------------\n";
    time_t begin = clock();

    gridStrategy->initalNodes(grid);

    time_t end = clock();
    real time = (real)(real(end - begin) / CLOCKS_PER_SEC);
    *logging::out << logging::Logger::INTERMEDIATE << "Time initial field: " + SSTR(time / 1000) + "sec\n";
    *logging::out << logging::Logger::INTERMEDIATE << "-------------------------------------------\n";

    return grid;
}

HOST Grid::Grid()
{
    //printf("Constructor\n");
    //this->print();
};

HOST Grid::~Grid()
{
    //printf("Destructor\n");
    //this->print();
};

HOSTDEVICE uint Grid::getSize() const
{
    return this->size;
}

HOSTDEVICE uint Grid::getReducedSize() const
{
    return this->reducedSize;
}

HOST void Grid::mesh(Geometry &geometry)
{
    clock_t begin = clock();

    gridStrategy->mesh(shared_from_this(), geometry);

    clock_t end = clock();
    real time = (real)(real(end - begin) / CLOCKS_PER_SEC);

    *logging::out << logging::Logger::INTERMEDIATE << "time grid generation: " + SSTR(time) + "s\n";
}

HOST void Grid::freeMemory()
{
    gridStrategy->freeMemory(shared_from_this());
}

HOST void Grid::removeOverlapNodes(SPtr<Grid> finerGrid)
{
    gridStrategy->removeOverlapNodes(shared_from_this(), finerGrid);
}

HOSTDEVICE void Grid::createGridInterface(uint index, const Grid& finerGrid)
{
    this->findGridInterface(index, finerGrid);
    this->setOverlapNodeToInvalid(index, finerGrid);
}

HOSTDEVICE void Grid::findGridInterface(uint index, const Grid& finerGrid)
{
    gridInterface->findCF(index, this, &finerGrid);
    gridInterface->findFC(index, this, &finerGrid);
}

HOSTDEVICE void Grid::setOverlapNodeToInvalid(uint index, const Grid& finerGrid)
{
    if (this->isInside(index, finerGrid))
        this->setFieldEntryToInvalid(index);
}

HOSTDEVICE bool Grid::isInside(uint index, const Grid& finerGrid) const
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    const real overlapWithStopper = 3 * this->delta;
    const real overlap = 2 * this->delta;

    return 
        (x > finerGrid.startX + overlapWithStopper && x < finerGrid.endX - overlap) &&
        (y > finerGrid.startY + overlapWithStopper && y < finerGrid.endY - overlap) &&
        (z > finerGrid.startZ + overlapWithStopper && z < finerGrid.endZ - overlap);
}

HOST void Grid::setPeriodicity(bool periodicityX, bool periodicityY, bool periodicityZ)
{
    this->periodicityX = periodicityX;
    this->periodicityY = periodicityY;
    this->periodicityZ = periodicityZ;
}

HOSTDEVICE bool Grid::isFluid(uint index) const
{
    return field[index] == FLUID;
}

HOSTDEVICE bool Grid::isSolid(uint index) const
{
    return field[index] == SOLID;
}

HOSTDEVICE bool Grid::isInvalid(uint index) const
{
    return field[index] == INVALID_NODE;
}

HOSTDEVICE bool Grid::isQ(uint index) const
{
    return field[index] == Q;
}

HOSTDEVICE bool Grid::isRb(uint index) const
{
    return field[index] == VELOCITY || field[index] == PRESSURE || field[index] == NOSLIP || field[index] == SOLID;
}

HOSTDEVICE void Grid::setFieldEntryToFluid(uint index)
{
    this->field[index] = FLUID;
}

HOSTDEVICE void Grid::setFieldEntryToSolid(uint index)
{
    this->field[index] = SOLID;
}

HOST void Grid::setFieldEntryToInvalid(uint index)
{
    this->field[index] = INVALID_NODE;
}

HOSTDEVICE void Grid::setFieldEntry(const Vertex &v, char val)
{
    this->field[transCoordToIndex(v)] = val;
}

HOSTDEVICE char Grid::getFieldEntry(const Vertex &v) const
{
    return this->field[transCoordToIndex(v)];
}

HOSTDEVICE int Grid::transCoordToIndex(const real &x, const real &y, const real &z) const
{
    return transCoordToIndex(Vertex(x,y,z));
}

HOSTDEVICE int Grid::transCoordToIndex(const Vertex &v) const
{
    const int x = int((v.x - startX) / delta);
    const int y = int((v.y - startY) / delta);
    const int z = int((v.z - startZ) / delta);

#ifdef DEBUG
    if (x < 0 || y < 0 || z < 0 || uint(x) >= nx || uint(y) >= ny || uint(z) >= nz)
    {
        printf(
            "Function: transCoordToIndex. Coordinates are out of range and cannot calculate the index. Exit Program!\n");
        /* exit(1);*/
    }
#endif

    return x + nx * (y + ny * z);
}

HOSTDEVICE void Grid::transIndexToCoords(const int index, real &x, real &y, real &z) const
{
#ifdef DEBUG
    if (index < 0 || index >= (int)size)
    {
        printf("Function: transIndexToCoords. Grid Index: %d, size: %d. Exit Program!\n", index, size); /*exit(1);*/ 
    }
#endif
    x = index % nx;
    y = (index / nx) % ny;
    z = ((index / nx) / ny) % nz;

    x = (x * delta) + startX;
    y = (y * delta) + startY;
    z = (z * delta) + startZ;
}


HOSTDEVICE void Grid::setDebugPoint(const Vertex &point, const int pointValue)
{
    if (getFieldEntry(point) == INVALID_NODE && pointValue == SOLID)
        setFieldEntry(point, pointValue);

    if (getFieldEntry(point) != SOLID && getFieldEntry(point) != Q && getFieldEntry(point) != INVALID_NODE && pointValue
        != 3 && pointValue != 2)
        setFieldEntry(point, pointValue);
}

HOSTDEVICE bool Grid::isOutOfRange(const Vertex &v) const
{
    return v.x < startX || v.y < startY || v.z < startZ || v.x > endX || v.y > endY || v.z > endZ;
}

HOSTDEVICE void Grid::meshTriangle(const Triangle &triangle)
{
    BoundingBox<real> box = BoundingBox<real>::makeRealNodeBox(triangle, delta);

    for (real x = box.minX; x <= box.maxX; x += delta)
    {
        for (real y = box.minY; y <= box.maxY; y += delta)
        {
            for (real z = box.minZ; z <= box.maxZ; z += delta)
            {
                Vertex point(x, y, z);
                if (isOutOfRange(point))
                    continue;
                const int value = triangle.isUnderFace(point);
                setDebugPoint(point, value);

                if (value == Q)
                    calculateQs(point, triangle);
            }
        }
    }
}

HOSTDEVICE void Grid::calculateQs(const Vertex &point, const Triangle &triangle)
{
    Vertex pointOnTriangle, direction;
    //VertexInteger solid_node;
    real subdistance;
    int error;
    for (int i = distribution.dir_start; i <= distribution.dir_end; i++)
    {
#if defined(__CUDA_ARCH__)
        direction = Vertex(DIRECTIONS[i][0], DIRECTIONS[i][1], DIRECTIONS[i][2]);
#else
        direction = Vertex(real(distribution.dirs[i * DIMENSION + 0]), real(distribution.dirs[i * DIMENSION + 1]),
                           real(distribution.dirs[i * DIMENSION + 2]));
#endif

        error = triangle.getTriangleIntersection(point, direction, pointOnTriangle, subdistance);

        if (error != 0 && subdistance <= 1.0f)
        {
            //solid_node = VertexInteger(actualPoint.x + direction.x, actualPoint.y + direction.y, actualPoint.z + direction.z);
            distribution.f[i*size + transCoordToIndex(point)] = subdistance;
            //printf("Q%d %d: %2.8f \n", i, grid.transCoordToIndex(actualPoint), grid.d.f[index]);
        }
    }
}

HOSTDEVICE void Grid::setNeighborIndices(const int &index)
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    real neighborX, neighborY, neighborZ;
    this->getNeighborCoords(neighborX, neighborY, neighborZ, x, y, z);

    neighborIndexX[index] = uint(transCoordToIndex(neighborX, y, z));
    neighborIndexY[index] = uint(transCoordToIndex(x, neighborY, z));
    neighborIndexZ[index] = uint(transCoordToIndex(x, y, neighborZ));
}

HOSTDEVICE void Grid::setInvalidNode(const int &index, bool &invalidNodeFound)
{
    if (isSolid(index))
        return;

    if (field[index] != INVALID_NODE && isNeighborInvalid(index))
    {
        field[index] = INVALID_NODE;
        invalidNodeFound = true;
    }
}

HOSTDEVICE bool Grid::isNeighborInvalid(const int &index)
{
    return (field[neighborIndexX[index]] == INVALID_NODE || field[neighborIndexY[index]] == INVALID_NODE || field[
        neighborIndexZ[index]] == INVALID_NODE);
}

HOSTDEVICE void Grid::findNeighborIndex(int index)
{
    if (this->matrixIndex[index] == -1)
    {
        this->neighborIndexX[index] = -1;
        this->neighborIndexY[index] = -1;
        this->neighborIndexZ[index] = -1;
        return;
    }

    if(this->isOverlapStopper(index) || isEndOfGridStopper(index))
    {
        this->setFieldEntryToSolid(index);
        this->setStopperNeighborCoords(index);
        return;
    }

    real x, y, z;
    this->transIndexToCoords(index, x, y, z);
    real neighborXCoord, neighborYCoord, neighborZCoord;
    getNeighborCoords(neighborXCoord, neighborYCoord, neighborZCoord, x, y, z);
    this->neighborIndexX[index] = getNeighborIndex(neighborXCoord, y, z);
    this->neighborIndexY[index] = getNeighborIndex(x, neighborYCoord, z);
    this->neighborIndexZ[index] = getNeighborIndex(x, y, neighborZCoord);
}

HOSTDEVICE bool Grid::isOverlapStopper(uint index) const
{
    return this->isFluid(index) && nodeInNextCellIsInvalid(index);
}


HOSTDEVICE bool Grid::nodeInNextCellIsInvalid(int index) const
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    real neighborX, neighborY, neighborZ;
    this->getNeighborCoords(neighborX, neighborY, neighborZ, x, y, z);

    const uint indexX = (uint)transCoordToIndex(neighborX, y, z);
    const uint indexY = (uint)transCoordToIndex(x, neighborY, z);
    const uint indexZ = (uint)transCoordToIndex(x, y, neighborZ);
     
    const uint indexXY = (uint)transCoordToIndex(neighborX, neighborY, z);
    const uint indexYZ = (uint)transCoordToIndex(x, neighborY, neighborZ);
    const uint indexXZ = (uint)transCoordToIndex(neighborX, y, neighborZ);
     
    const uint indexXYZ = (uint)transCoordToIndex(neighborX, neighborY, neighborZ);

    const bool isInvalidNeighborX = this->isInvalid(indexX);
    const bool isInvalidNeighborY = this->isInvalid(indexY);
    const bool isInvalidNeighborXY  = this->isInvalid(indexXY);
    const bool isInvalidNeighborZ   = this->isInvalid(indexZ);
    const bool isInvalidNeighborYZ  = this->isInvalid(indexYZ);
    const bool isInvalidNeighborXZ  = this->isInvalid(indexXZ);
    const bool isInvalidNeighborXYZ = this->isInvalid(indexXYZ);

    return isInvalidNeighborX || isInvalidNeighborY || isInvalidNeighborXY || isInvalidNeighborZ || isInvalidNeighborYZ
        || isInvalidNeighborXZ || isInvalidNeighborXYZ;
}

HOSTDEVICE int Grid::getNeighborIndex(const real &expectedX, const real &expectedY, const real &expectedZ) const
{
    const int neighborIndex = transCoordToIndex(expectedX, expectedY, expectedZ);
    return matrixIndex[neighborIndex];
}

//void findForGridInterfaceNewIndex(GridInterface::Interface interface, uint interfaceIndex, int* indices)
//{
//    const uint oldIndex = interface.coarse[interfaceIndex];
//    const uint newIndex = indices[oldIndex];
//    interface.coarse[interfaceIndex] = newIndex;
//}

HOST void Grid::findForGridInterfaceNewIndexCF(uint index)
{
    const uint oldIndex = gridInterface->cf.coarse[index];
    const uint newIndex = matrixIndex[oldIndex];
    gridInterface->cf.coarse[index] = newIndex;
}

HOST void Grid::findForGridInterfaceNewIndexFC(uint index)
{
    const uint oldIndex = gridInterface->fc.coarse[index];
    const uint newIndex = matrixIndex[oldIndex];
    gridInterface->fc.coarse[index] = newIndex;
}


HOST void Grid::removeInvalidNodes()
{
    int removedNodes = 0;
    int newIndex = 0;
    for (uint index = 0; index < size; index++)
    {
        if (this->isInvalid(index))
        {
            matrixIndex[index] = -1;
            removedNodes++;
        } else
        {
            matrixIndex[index] = newIndex;
            newIndex++;
        }
    }
    reducedSize = size - removedNodes;
    printf("new size coords: %d , delete nodes: %d\n", reducedSize, removedNodes);
}

HOSTDEVICE void Grid::getNeighborCoords(real &neighborX, real &neighborY, real &neighborZ, real x, real y, real z) const
{
    neighborX = getNeighhborCoord(periodicityX, x, startX, endX);
    neighborY = getNeighhborCoord(periodicityY, y, startY, endY);
    neighborZ = getNeighhborCoord(periodicityZ, z, startZ, endZ);
}

HOSTDEVICE real Grid::getNeighhborCoord(bool periodicity, real actualCoord, real startCoord, real endCoord) const
{
    if (periodicity)
        return CudaMath::lessEqual(actualCoord + delta, endCoord) ? actualCoord + delta : startCoord;
    else
        return actualCoord + delta;
}

HOSTDEVICE void Grid::setStopperNeighborCoords(int index)
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    if (CudaMath::lessEqual(x + delta, endX + delta))
        neighborIndexX[index] = getNeighborIndex(x + delta, y, z);

    if (CudaMath::lessEqual(y + delta, endY + delta))
        neighborIndexY[index] = getNeighborIndex(x, y + delta, z);

    if (CudaMath::lessEqual(z + delta, endZ + delta))
        neighborIndexZ[index] = getNeighborIndex(x, y, z + delta);
}


HOSTDEVICE bool Grid::isEndOfGridStopper(uint index) const
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);
    return (x > this->endX || y > this->endY || z > this->endZ);
}


uint Grid::getNumberOfNodesCF() const
{
    if(this->gridInterface)
        return this->gridInterface->cf.numberOfEntries;
    return 0;
}

uint Grid::getNumberOfNodesFC() const
{
    if (this->gridInterface)
     return this->gridInterface->fc.numberOfEntries;
    return 0;
}

uint* Grid::getCF_coarse() const
{
    return this->gridInterface->cf.coarse;
}

uint* Grid::getCF_fine() const
{
    return this->gridInterface->cf.fine;
}

uint* Grid::getFC_coarse() const
{
    return this->gridInterface->fc.coarse;
}

uint* Grid::getFC_fine() const
{
    return this->gridInterface->fc.fine;
}

void Grid::getGridInterfaceIndices(uint* iCellCfc, uint* iCellCff, uint* iCellFcc, uint* iCellFcf) const
{
    setGridInterface(iCellCfc, this->gridInterface->cf.coarse, this->gridInterface->cf.numberOfEntries);
    setGridInterface(iCellCff, this->gridInterface->cf.fine, this->gridInterface->cf.numberOfEntries);
    setGridInterface(iCellFcc, this->gridInterface->fc.coarse, this->gridInterface->fc.numberOfEntries);
    setGridInterface(iCellFcf, this->gridInterface->fc.fine, this->gridInterface->fc.numberOfEntries);
}

void Grid::setGridInterface(uint* gridInterfaceList, const uint* oldGridInterfaceList, uint size)
{
    for (uint i = 0; i < size; i++)
        gridInterfaceList[i] = oldGridInterfaceList[i] + 1;
}



HOSTDEVICE void Grid::print() const
{
    printf("min: (%2.2f, %2.2f, %2.2f), max: (%2.2f, %2.2f, %2.2f), size: %d, delta: %2.2f\n", startX, startY, startZ,
           endX, endY, endZ, size, delta);
}

std::string Grid::toString() const
{
    std::ostringstream oss;
    oss << 
        "min: (" << startX << ", " << startY << ", " << startZ << 
        "), max: " << endX << ", " << endY << ", " << endZ <<
        "), size: " << reducedSize << ", delta: " << delta << "\n";
    return oss.str();
}


