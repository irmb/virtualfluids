#include "hip/hip_runtime.h"
#include "Grid.cuh"

#include "GridGenerator/global.h"
#include <stdio.h>
#include <time.h>

#include <sstream>
#include <algorithm>


#include <GridGenerator/utilities/math/CudaMath.cuh>

#include <GridGenerator/geometries/Vertex/Vertex.cuh>
#include <GridGenerator/geometries/Triangle/Triangle.cuh>
#include <GridGenerator/geometries/BoundingBox/BoundingBox.cuh>

#include <GridGenerator/grid/NodeValues.h>
#include <GridGenerator/grid/distributions/Distribution.h>

#include <GridGenerator/grid/GridStrategy/GridStrategy.h>
#include <utilities/logger/Logger.h>
#include "GridInterface.cuh"


CONSTANT int DIRECTIONS[DIR_END_MAX][DIMENSION];

HOST Grid::Grid(real startX, real startY, real startZ, real endX, real endY, real endZ, real delta, std::shared_ptr<GridStrategy> gridStrategy, Distribution &d) 
: startX(startX), startY(startY), startZ(startZ), endX(endX), endY(endY), endZ(endZ), delta(delta), gridStrategy(gridStrategy), d(d)
{
    const real length = endX - startX;
    const real width = endY - startY;
    const real height = endZ - startZ;

    nx = int((length + delta) / delta);
    ny = int((width  + delta) / delta);
    nz = int((height + delta) / delta);

    this->size = nx * ny * nz;
    this->reducedSize = size;
}

HOST SPtr<Grid> Grid::makeShared(real startX, real startY, real startZ, real endX, real endY, real endZ, real delta, std::shared_ptr<GridStrategy> gridStrategy, Distribution &d)
{
    SPtr<Grid> grid(new Grid(startX, startY, startZ, endX, endY, endZ, delta, gridStrategy, d));

    gridStrategy->allocateGridMemory(grid);
    *logging::out << logging::Logger::LOW << "-------------------------------------------\n";
    *logging::out << logging::Logger::LOW << "Initial field with fluid. \n";
    *logging::out << logging::Logger::LOW << "-------------------------------------------\n";
    time_t begin = clock();
    gridStrategy->initalNodes(grid);

    time_t end = clock();
    real time = (real)(real(end - begin) / CLOCKS_PER_SEC);
    *logging::out << logging::Logger::INTERMEDIATE << "Time initial field: " + SSTR(time / 1000) + "sec\n";
    *logging::out << logging::Logger::INTERMEDIATE << "-------------------------------------------\n";

    return grid;
}

HOST Grid::Grid(){};


HOST Grid::Grid(char *field, int startX, int startY, int startZ, int eX, int eY, int eZ, Distribution &d)
    : field(field), startX(startX), startY(startY), startZ(startZ), endX(eX), endY(eY), endZ(eZ), d(d)
{
    nx = eX;
    ny = eY;
    nz = eZ;
    this->size = eX * eY * eZ;
    this->reducedSize = size;
}

HOST void Grid::mesh(Geometry &geometry)
{
    clock_t begin = clock();

    gridStrategy->mesh(shared_from_this(), geometry);

    clock_t end = clock();
    real time = (real)(real(end - begin) / CLOCKS_PER_SEC);

    *logging::out << logging::Logger::INTERMEDIATE << "time grid generation: " + SSTR(time) + "s\n";
}

HOST void Grid::freeMemory()
{
    gridStrategy->freeMemory(shared_from_this());
}

HOST void Grid::removeOverlapNodes(SPtr<Grid> finerGrid)
{
    gridStrategy->removeOverlapNodes(shared_from_this(), finerGrid);
}

HOSTDEVICE void Grid::setOverlapNodeToInvalid(uint index, const Grid& finerGrid)
{
    if (this->isInside(index, finerGrid))
        this->setFieldEntryToInvalid(index);
}

HOSTDEVICE bool Grid::isInside(uint index, const Grid& finerGrid)
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    const real overlapWithStopper = 3 * this->delta;
    const real overlap = 2 * this->delta;

    gridInterface->findCF(index, this, &finerGrid);
    gridInterface->findFC(index, this, &finerGrid);

    return 
        (x > finerGrid.startX + overlapWithStopper && x < finerGrid.endX - overlap) &&
        (y > finerGrid.startY + overlapWithStopper && y < finerGrid.endY - overlap) &&
        (z > finerGrid.startZ + overlapWithStopper && z < finerGrid.endZ - overlap);
}

HOSTDEVICE bool Grid::isFluid(uint index) const
{
    return field[index] == FLUID;
}

HOSTDEVICE bool Grid::isSolid(uint index) const
{
    return field[index] == SOLID;
}

HOSTDEVICE bool Grid::isInvalid(uint index) const
{
    return field[index] == INVALID_NODE;
}

HOSTDEVICE bool Grid::isQ(uint index) const
{
    return field[index] == Q;
}

HOSTDEVICE  bool Grid::isRb(uint index) const
{
    return field[index] == VELOCITY || field[index] == PRESSURE || field[index] == NOSLIP || field[index] == SOLID;
}

HOSTDEVICE void Grid::setFieldEntryToFluid(uint index)
{
	this->field[index] = FLUID;
}

HOSTDEVICE void Grid::setFieldEntryToSolid(uint index)
{
	this->field[index] = SOLID;
}

HOST void Grid::setFieldEntryToInvalid(uint index)
{
    this->field[index] = INVALID_NODE;
}

HOSTDEVICE void Grid::setFieldEntry(const Vertex &v, char val)
{
    this->field[transCoordToIndex(v)] = val;
}

HOSTDEVICE char Grid::getFieldEntry(const Vertex &v) const
{
    return this->field[transCoordToIndex(v)];
}

HOSTDEVICE int Grid::transCoordToIndex(const real &x, const real &y, const real &z) const
{
	return transCoordToIndex(Vertex(x,y,z));
}

HOSTDEVICE int Grid::transCoordToIndex(const Vertex &v) const
{
#ifdef DEBUG
	if (isOutOfRange(v))
	{ printf("Function: transCoordToIndex. Coordinates are out of range and cannot calculate the index. Exit Program!\n");/* exit(1);*/ };
#endif
    int x = (int)((v.x - startX) / delta);
    int y = (int)((v.y - startY) / delta);
    int z = (int)((v.z - startZ) / delta);

	return x + nx * (y + ny * z);
}

HOSTDEVICE void Grid::transIndexToCoords(const int index, real &x, real &y, real &z) const
{
#ifdef DEBUG
	if (index < 0 || index >= (int)size)
	{
        printf("Function: transIndexToCoords. Grid Index: %d, size: %d. Exit Program!\n", index, size); /*exit(1);*/ 
    };
#endif
    x = index % nx;
    y = (index / nx) % ny;
    z = ((index / nx) / ny) % nz;

    x = (x * delta) + startX;
    y = (y * delta) + startY;
    z = (z * delta) + startZ;
}

char* Grid::toString(const char* name) const
{
    std::stringstream ss;
    ss << "\n" << name << " " << nx << " " << ny << " " << nz;
    return strdup(ss.str().c_str());
}

HOSTDEVICE void Grid::print() const
{
    printf("min: (%2.2f, (%2.2f, %2.2f), max: (%2.2f, %2.2f, %2.2f), size: %d, delta: %2.2f\n", startX, startY, startZ, endX, endY, endZ, size, delta);
}

HOSTDEVICE void Grid::setDebugPoint(const Vertex &point, const int pointValue)
{
    if (getFieldEntry(point) == INVALID_NODE && pointValue == SOLID)
        setFieldEntry(point, pointValue);

	if (getFieldEntry(point) != SOLID && getFieldEntry(point) != Q && getFieldEntry(point) != INVALID_NODE && pointValue != 3 && pointValue != 2)
		setFieldEntry(point, pointValue);
}

HOSTDEVICE bool Grid::isOutOfRange(const Vertex &v) const
{
	return v.x < startX || v.y < startY || v.z < startZ || v.x > endX || v.y > endY || v.z > endZ;
}

HOSTDEVICE void Grid::meshTriangleExact(const Triangle &triangle)
{
    BoundingBox<real> box = BoundingBox<real>::makeRealNodeBox(triangle, delta);

    for (real x = box.minX; x <= box.maxX; x += delta) {
        for (real y = box.minY; y <= box.maxY; y += delta) {
            for (real z = box.minZ; z <= box.maxZ; z += delta) {
                Vertex point(x, y, z);
                if (isOutOfRange(point))
                    continue;
                const int value = triangle.isUnderFace(point);
                setDebugPoint(point, value);

                if (value == Q)
                    calculateQs(point, triangle);
            }
        }
    }
}

HOSTDEVICE void Grid::meshTriangle(const Triangle &triangle)
{
	int x, y, z;
	Vertex point;

	BoundingBox<int> box = BoundingBox<int>::makeNodeBox(triangle);

	for (x = box.minX; x <= box.maxX; x++) {
		for (y = box.minY; y <= box.maxY; y++) {
			for (z = box.minZ; z <= box.maxZ; z++) {
				point = Vertex((real)x, (real)y, (real)z);
				if (isOutOfRange(point))
					continue;
                int value = triangle.isUnderFace(point);
                setDebugPoint(point, value);

                if (value == Q)
                    calculateQs(point, triangle);
			}
		}
	}
}

HOSTDEVICE void Grid::calculateQs(const Vertex &point, const Triangle &triangle)
{
	Vertex pointOnTriangle, direction;
	//VertexInteger solid_node;
	real subdistance;
	int error;
	for (int i = d.dir_start; i <= d.dir_end; i++) {
	#if defined(__CUDA_ARCH__)
        direction = Vertex(DIRECTIONS[i][0], DIRECTIONS[i][1], DIRECTIONS[i][2]);
	#else
        direction = Vertex((real)d.dirs[i * DIMENSION + 0], (real)d.dirs[i * DIMENSION + 1], (real)d.dirs[i * DIMENSION + 2]);
	#endif

        error = triangle.getTriangleIntersection(point, direction, pointOnTriangle, subdistance);

		if (error != 0 && subdistance <= 1.0f) {
			//solid_node = VertexInteger(actualPoint.x + direction.x, actualPoint.y + direction.y, actualPoint.z + direction.z);
			d.f[i*size + transCoordToIndex(point)] = subdistance;
			//printf("Q%d %d: %2.8f \n", i, grid.transCoordToIndex(actualPoint), grid.d.f[index]);
		}
	}
}

HOSTDEVICE void Grid::setNeighborIndices(const int &index)
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    real neighborX, neighborY, neighborZ;
    this->getNeighborCoords(neighborX, neighborY, neighborZ, x, y, z);

	neighborIndexX[index] = (uint) transCoordToIndex(neighborX, y, z);
    neighborIndexY[index] = (uint) transCoordToIndex(x, neighborY, z);
    neighborIndexZ[index] = (uint) transCoordToIndex(x, y, neighborZ);

	//if (grid.isRB(index)) {
    //if (neighborX == 0) neighborIndexX[index] = 0;
    //if (neighborY == 0) neighborIndexY[index] = 0;
    //if (neighborZ == 0) neighborIndexZ[index] = 0;
	//}
}

HOSTDEVICE void Grid::setInvalidNode(const int &index, bool &invalidNodeFound)
{
    if (isSolid(index))
        return;

    if (field[index] != INVALID_NODE && isNeighborInvalid(index))
    {
        field[index] = INVALID_NODE;
        invalidNodeFound = true;
    }
}


HOSTDEVICE bool Grid::isNeighborInvalid(const int &index)
{
    return (field[neighborIndexX[index]] == INVALID_NODE || field[neighborIndexY[index]] == INVALID_NODE || field[neighborIndexZ[index]] == INVALID_NODE);
}

HOSTDEVICE void Grid::findNeighborIndex(int index)
{
    if (this->matrixIndex[index] == -1)
    {
        this->neighborIndexX[index] = -1;
        this->neighborIndexY[index] = -1;
        this->neighborIndexZ[index] = -1;
        return;
    }

    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    if(this->isOverlapStopper(index))
    {
        this->setFieldEntryToSolid(index);
        this->neighborIndexX[index] = -1;
        this->neighborIndexY[index] = -1;
        this->neighborIndexZ[index] = -1;
        return;
    }

    real neighborXCoord, neighborYCoord, neighborZCoord;
    getNeighborCoords(neighborXCoord, neighborYCoord, neighborZCoord, x, y, z);
    this->neighborIndexX[index] = getNeighborIndex(/*index, this->neighborIndexX[nodeIndex],*/ neighborXCoord, y, z);
    this->neighborIndexY[index] = getNeighborIndex(/*index, this->neighborIndexY[nodeIndex],*/ x, neighborYCoord, z);
    this->neighborIndexZ[index] = getNeighborIndex(/*index, this->neighborIndexZ[nodeIndex],*/ x, y, neighborZCoord);
}

HOSTDEVICE bool Grid::isOverlapStopper(uint index) const
{
    return this->isFluid(index) && nodeInNextCellIsInvalid(index);
}


HOSTDEVICE bool Grid::nodeInNextCellIsInvalid(int index) const
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    real neighborX, neighborY, neighborZ;
    this->getNeighborCoords(neighborX, neighborY, neighborZ, x, y, z);

    const uint indexX = (uint)transCoordToIndex(neighborX, y, z);
    const uint indexY = (uint)transCoordToIndex(x, neighborY, z);
    const uint indexZ = (uint)transCoordToIndex(x, y, neighborZ);
     
    const uint indexXY = (uint)transCoordToIndex(neighborX, neighborY, z);
    const uint indexYZ = (uint)transCoordToIndex(x, neighborY, neighborZ);
    const uint indexXZ = (uint)transCoordToIndex(neighborX, y, neighborZ);
     
    const uint indexXYZ = (uint)transCoordToIndex(neighborX, neighborY, neighborZ);

    const bool isInvalidNeighborX = this->isInvalid(indexX);
    const bool isInvalidNeighborY = this->isInvalid(indexY);
    const bool isInvalidNeighborXY  = this->isInvalid(indexXY);
    const bool isInvalidNeighborZ   = this->isInvalid(indexZ);
    const bool isInvalidNeighborYZ  = this->isInvalid(indexYZ);
    const bool isInvalidNeighborXZ  = this->isInvalid(indexXZ);
    const bool isInvalidNeighborXYZ = this->isInvalid(indexXYZ);

    return isInvalidNeighborX || isInvalidNeighborY || isInvalidNeighborXY || isInvalidNeighborZ || isInvalidNeighborYZ || isInvalidNeighborXZ || isInvalidNeighborXYZ;
}

HOSTDEVICE int Grid::getNeighborIndex(/*const int &nodeIndex, const int &neighborIndex, */const real &expectedX, const real &expectedY, const real &expectedZ)
{

    int neighborIndex = transCoordToIndex(expectedX, expectedY, expectedZ);
    return matrixIndex[neighborIndex];

    //int newNeighborIndex = neighborIndex;
    //while (newNeighborIndex >= (int)this->reducedSize)
    //    newNeighborIndex--;

    //if (newNeighborIndex >= 0)
    //{
    //    real neighborX, neighborY, neighborZ;
    //    this->transIndexToCoords(this->matrixIndex[newNeighborIndex], neighborX, neighborY, neighborZ);
    //    while (!(neighborX == expectedX && neighborY == expectedY && neighborZ == expectedZ)) {
    //        newNeighborIndex--;
    //        //printf("expectedNeighborCoords:(%d, %d, %d), actualNeighborCoords:(%d,%d,%d), neighborIndex: %d\n", expectedX, expectedY, expectedZ,neighborX ,neighborY ,neighborZ, neighborIndex);
    //        this->transIndexToCoords(this->matrixIndex[newNeighborIndex], neighborX, neighborY, neighborZ);

    //        if (newNeighborIndex == nodeIndex)
    //            return -1;
    //    }
    //    return newNeighborIndex;
    //}
    //return -1;
}


HOST void Grid::removeInvalidNodes()
{
    int removedNodes = 0;
    int newIndex = 0;
    for (uint index = 0; index < size; index++)
    {
        if (this->isInvalid(index))
        {
            matrixIndex[index] = -1;
            removedNodes++;
        } else
        {
            matrixIndex[index] = newIndex;
            newIndex++;
        }
    }
    reducedSize = size - removedNodes;
    printf("new size coords: %zd , delete nodes: %zd\n", reducedSize, removedNodes);


    //std::vector<uint> stl_vector(size);
    //stl_vector.assign(this->matrixIndex, this->matrixIndex + this->size);

    //int oldsize = (int)stl_vector.size();
    //printf("size coords: %d \n", oldsize);
    //std::vector<uint>::iterator end_vaild = std::remove_if(stl_vector.begin(), stl_vector.end(), [this](const uint &index)
    //{
    //    return this->field[index] == INVALID_NODE;
    //});

    //stl_vector.erase(end_vaild, stl_vector.end());
    //printf("new size coords: %zd , delete nodes: %zd\n", stl_vector.size(), oldsize - stl_vector.size());

    //uint *indices_reduced = new uint[stl_vector.size()];
    //for (size_t i = 0; i < stl_vector.size(); i++)
    //    indices_reduced[i] = stl_vector[i];
    //
    //this->reducedSize = (int)stl_vector.size();
    //delete[]this->matrixIndex;
    //this->matrixIndex = indices_reduced;
}

HOSTDEVICE void Grid::getNeighborCoords(real &neighborX, real &neighborY, real &neighborZ, const real x, const real y, const real z) const
{
	neighborX = x + delta < endX ? x + delta : startX;
    neighborY = y + delta < endY ? y + delta : startY;
    neighborZ = z + delta < endZ ? z + delta : startZ;
}

HOSTDEVICE bool Grid::isStopper(int index) const
{
    return isSolid(index) && previousCellHasFluid(index);
}



HOSTDEVICE bool Grid::previousCellHasFluid(int index) const
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    real previousX = x - delta >= 0 ? x - delta : this->endX;
    real previousY = y - delta >= 0 ? y - delta : this->endY;
    real previousZ = z - delta >= 0 ? z - delta : this->endZ;

    int indexpreviousX   = this->transCoordToIndex(previousX, y, z);
    int indexpreviousY   = this->transCoordToIndex(x, previousY, z);
    int indexpreviousXY  = this->transCoordToIndex(previousX, previousY, z);
    int indexpreviousZ   = this->transCoordToIndex(x, y, previousZ);
    int indexpreviousZX  = this->transCoordToIndex(previousX, y, previousZ);
    int indexpreviousZY  = this->transCoordToIndex(x, previousY, previousZ);
    int indexpreviousZYX = this->transCoordToIndex(previousX, previousY, previousZ);

    return (!this->isSolid(indexpreviousX) || !this->isSolid(indexpreviousY) || !this->isSolid(indexpreviousZ)
        || !this->isSolid(indexpreviousZYX) || !this->isSolid(indexpreviousXY) || !this->isSolid(indexpreviousZY) || !this->isSolid(indexpreviousZX));
}
