#include "hip/hip_runtime.h"
#include "GridImp.h"

#include <stdio.h>
#include <time.h>
#include <iostream>
#include <omp.h>
#include <sstream>

#include "global.h"

#include "geometries/Object.h"
#include "geometries/Vertex/Vertex.h"
#include "geometries/Triangle/Triangle.h"
#include "geometries/TriangularMesh/TriangularMesh.h"
#include "geometries/TriangularMesh/TriangularMeshStrategy.h"
#include "geometries/BoundingBox/BoundingBox.h"

#include "grid/GridStrategy/GridStrategy.h"
#include "grid/distributions/Distribution.h"
#include "grid/Field.h"
#include "grid/GridInterface.h"
#include "grid/NodeValues.h"

#include "io/GridVTKWriter/GridVTKWriter.h"

#include "utilities/communication.h"
#include "utilities/math/Math.h"

CONSTANT int DIRECTIONS[DIR_END_MAX][DIMENSION];


HOST GridImp::GridImp(Object* object, real startX, real startY, real startZ, real endX, real endY, real endZ, real delta, SPtr<GridStrategy> gridStrategy, Distribution distribution, uint level) 
            : object(object), 
    startX(startX),
    startY(startY),
    startZ(startZ),
    endX(endX),
    endY(endY),
    endZ(endZ),
    delta(delta),
    gridStrategy(gridStrategy),
    distribution(distribution),
    level(level),
    periodicityX(false),
    periodicityY(false),
    periodicityZ(false),
    enableFixRefinementIntoTheWall(false),
    gridInterface(nullptr),
    neighborIndexX(nullptr),
    neighborIndexY(nullptr),
    neighborIndexZ(nullptr),
    neighborIndexNegative(nullptr),
    sparseIndices(nullptr),
    qIndices(nullptr),
    qValues(nullptr),
    qPatches(nullptr),
    innerRegionFromFinerGrid(false),
    numberOfLayers(0),
    qComputationStage(qComputationStageType::ComputeQs)
{
    initalNumberOfNodesAndSize();
}

HOST SPtr<GridImp> GridImp::makeShared(Object* object, real startX, real startY, real startZ, real endX, real endY, real endZ, real delta, SPtr<GridStrategy> gridStrategy, Distribution d, uint level)
{
    SPtr<GridImp> grid(new GridImp(object, startX, startY, startZ, endX, endY, endZ, delta, gridStrategy, d, level));
    return grid;
}


void GridImp::initalNumberOfNodesAndSize()
{
    const real length = endX - startX;
    const real width = endY - startY;
    const real height = endZ - startZ;

    nx = lround((length + delta) / delta);
    ny = lround((width + delta) / delta);
    nz = lround((height + delta) / delta);

    this->size = nx * ny * nz;
    this->sparseSize = size;
    distribution.setSize(size);

	this->numberOfSolidBoundaryNodes = 0;
}

HOST void GridImp::inital(const SPtr<Grid> fineGrid, uint numberOfLayers)
{
    field = Field(gridStrategy, size);
    field.allocateMemory();
    gridStrategy->allocateGridMemory(shared_from_this());
    
    *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start initalNodesToOutOfGrid()\n";
    gridStrategy->initalNodesToOutOfGrid(shared_from_this());
    
    if( this->innerRegionFromFinerGrid ){
        *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start setInnerBasedOnFinerGrid()\n";
        this->setInnerBasedOnFinerGrid(fineGrid);
    }
    else{
        *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start findInnerNodes()\n";
        this->object->findInnerNodes( shared_from_this() );
    }

    *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start addOverlap()\n";
    this->addOverlap();
    
    *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start fixOddCells()\n";
    gridStrategy->fixOddCells( shared_from_this() );
    
    if( enableFixRefinementIntoTheWall )
    {
        *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start fixRefinementIntoWall()\n";
        gridStrategy->fixRefinementIntoWall(shared_from_this());
    }
    
    *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start findEndOfGridStopperNodes()\n";
	gridStrategy->findEndOfGridStopperNodes(shared_from_this());

    *logging::out << logging::Logger::INFO_INTERMEDIATE
        << "Grid created: " << "from (" << this->startX << ", " << this->startY << ", " << this->startZ << ") to (" << this->endX << ", " << this->endY << ", " << this->endZ << ")\n"
        << "nodes: " << this->nx << " x " << this->ny << " x " << this->nz << " = " << this->size << "\n";
}

HOST void GridImp::setOddStart(bool xOddStart, bool yOddStart, bool zOddStart)
{
    this->xOddStart = xOddStart;
    this->yOddStart = yOddStart;
    this->zOddStart = zOddStart;
}

HOSTDEVICE void GridImp::initalNodeToOutOfGrid(uint index)
{
    this->field.setFieldEntryToInvalidOutOfGrid(index);
}

HOST void GridImp::freeMemory()
{
    gridStrategy->freeMemory(shared_from_this());
}

HOST GridImp::GridImp()
{
    //printf("Constructor\n");
    //this->print();
}

HOST GridImp::~GridImp()
{
    //printf("Destructor\n");
    //this->print();
}

HOSTDEVICE void GridImp::findInnerNode(uint index)
{
    this->sparseIndices[index] = index;

    if( this->level != 0 ){
        const Cell cell = getOddCellFromIndex(index);
        if (isInside(cell))
            this->field.setFieldEntryToFluid(index);
    }
    else{
        real x, y, z;
        this->transIndexToCoords(index, x, y, z);
        const uint xIndex = getXIndex(x);
        const uint yIndex = getYIndex(y);
        const uint zIndex = getZIndex(z);

        if( xIndex != 0 && xIndex != this->nx-1 &&
            yIndex != 0 && yIndex != this->ny-1 &&
            zIndex != 0 && zIndex != this->nz-1 )
            this->field.setFieldEntryToFluid(index);
    }
}

bool GridImp::isInside(const Cell& cell) const
{
    return object->isCellInObject(cell);
}

////TODO: check where the fine grid starts (0.25 or 0.75) and if even or odd-cell is needed
// Cell numbering:
//       even start                            odd start
//    +---------+                           +---------+
//    |       +-----+-----+-----+           | +-----+-----+-----+
//    |       | |   |     |     |           | |     | |   |     |
//    |       +-----+-----+-----+           | +-----+-----+-----+
//    +---------+                           +---------+
//               0     1     2                   0     1     2
//              even      even                        even     
//                   odd                        odd         odd
//
HOSTDEVICE Cell GridImp::getOddCellFromIndex(uint index) const
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    const uint xIndex = getXIndex(x);
    const uint yIndex = getYIndex(y);
    const uint zIndex = getZIndex(z);

    real xCellStart;
    if( this->xOddStart ) xCellStart = xIndex % 2 != 0 ? x - this->delta : x;
    else                  xCellStart = xIndex % 2 != 0 ? x               : x - this->delta;

    real yCellStart;
    if( this->yOddStart ) yCellStart = yIndex % 2 != 0 ? y - this->delta : y;
    else                  yCellStart = yIndex % 2 != 0 ? y               : y - this->delta;

    real zCellStart;
    if( this->zOddStart ) zCellStart = zIndex % 2 != 0 ? z - this->delta : z;
    else                  zCellStart = zIndex % 2 != 0 ? z               : z - this->delta;

    return Cell(xCellStart, yCellStart, zCellStart, delta);
}

HOSTDEVICE void GridImp::setInnerBasedOnFinerGrid(const SPtr<Grid> fineGrid)
{
    for( uint index = 0; index < this->size; index++ ){

        real x, y, z;
        this->transIndexToCoords(index, x, y, z);

        uint childIndex[8];

        childIndex[0] = fineGrid->transCoordToIndex( x + 0.25 * this->delta, y + 0.25 * this->delta, z + 0.25 * this->delta );
        childIndex[1] = fineGrid->transCoordToIndex( x + 0.25 * this->delta, y + 0.25 * this->delta, z - 0.25 * this->delta );
        childIndex[2] = fineGrid->transCoordToIndex( x + 0.25 * this->delta, y - 0.25 * this->delta, z + 0.25 * this->delta );
        childIndex[3] = fineGrid->transCoordToIndex( x + 0.25 * this->delta, y - 0.25 * this->delta, z - 0.25 * this->delta );
        childIndex[4] = fineGrid->transCoordToIndex( x - 0.25 * this->delta, y + 0.25 * this->delta, z + 0.25 * this->delta );
        childIndex[5] = fineGrid->transCoordToIndex( x - 0.25 * this->delta, y + 0.25 * this->delta, z - 0.25 * this->delta );
        childIndex[6] = fineGrid->transCoordToIndex( x - 0.25 * this->delta, y - 0.25 * this->delta, z + 0.25 * this->delta );
        childIndex[7] = fineGrid->transCoordToIndex( x - 0.25 * this->delta, y - 0.25 * this->delta, z - 0.25 * this->delta );

        for( uint i = 0; i < 8; i++ ){
            if( childIndex[i] != INVALID_INDEX && fineGrid->getFieldEntry( childIndex[i] ) == FLUID ){
                this->setFieldEntry(index, FLUID);
                break;
            }
        }
    }
}

HOSTDEVICE void GridImp::addOverlap()
{
    for( uint layer = 0; layer < this->numberOfLayers; layer++ ){
        this->gridStrategy->addOverlap( shared_from_this() );
    }
}

HOSTDEVICE void GridImp::setOverlapTmp( uint index )
{
    if( this->field.is( index, INVALID_OUT_OF_GRID ) ){
        
        if( this->hasNeighborOfType(index, FLUID) ){
            this->field.setFieldEntry( index, OVERLAP_TMP );
        }
    }
}

HOSTDEVICE void GridImp::setOverlapFluid( uint index )
{
    if( this->field.is( index, OVERLAP_TMP ) ){
        this->field.setFieldEntry( index, FLUID );
    }
}

HOSTDEVICE void GridImp::fixRefinementIntoWall(uint xIndex, uint yIndex, uint zIndex, int dir)
{

    real x = this->startX + this->delta * xIndex;
    real y = this->startY + this->delta * yIndex;
    real z = this->startZ + this->delta * zIndex;

    uint index = this->transCoordToIndex(x, y, z);

    if( !this->xOddStart && ( dir == 1 || dir == -1 ) && ( xIndex % 2 == 1 || xIndex == 0 ) ) return;
    if( !this->yOddStart && ( dir == 2 || dir == -2 ) && ( yIndex % 2 == 1 || yIndex == 0 ) ) return;
    if( !this->zOddStart && ( dir == 3 || dir == -3 ) && ( zIndex % 2 == 1 || zIndex == 0 ) ) return;

    // Dont do this if inside of the domain
    if(  this->xOddStart && ( dir == 1 || dir == -1 ) && ( xIndex % 2 == 0 && xIndex != 0 ) ) return;
    if(  this->yOddStart && ( dir == 2 || dir == -2 ) && ( yIndex % 2 == 0 && yIndex != 0 ) ) return;
    if(  this->zOddStart && ( dir == 3 || dir == -3 ) && ( zIndex % 2 == 0 && zIndex != 0 ) ) return;
    
    //////////////////////////////////////////////////////////////////////////

    real dx, dy, dz;

    if      ( dir ==  1 ){ dx =   this->delta; dy = 0.0;           dz = 0.0;           }
    else if ( dir == -1 ){ dx = - this->delta; dy = 0.0;           dz = 0.0;           }
    else if ( dir ==  2 ){ dx = 0.0;           dy =   this->delta; dz = 0.0;           }
    else if ( dir == -2 ){ dx = 0.0;           dy = - this->delta; dz = 0.0;           }
    else if ( dir ==  3 ){ dx = 0.0;           dy = 0.0;           dz =   this->delta; }
    else if ( dir == -3 ){ dx = 0.0;           dy = 0.0;           dz = - this->delta; }

    //////////////////////////////////////////////////////////////////////////

    char type = this->field.getFieldEntry(index);

    char type2    = ( type == FLUID ) ? ( INVALID_OUT_OF_GRID ) : ( FLUID );
    uint distance = ( type == FLUID ) ? ( 9                   ) : ( 5     );

    bool allTypesAreTheSame = true;

    for( uint i = 1; i <= distance; i++ ){
        uint neighborIndex = this->transCoordToIndex(x + i * dx, y + i * dy, z + i * dz);

        if( neighborIndex != INVALID_INDEX && !this->field.is( neighborIndex, type ) )
            allTypesAreTheSame = false;
    }

    //////////////////////////////////////////////////////////////////////////

    if( allTypesAreTheSame )
        return;

    this->setFieldEntry(index, type2);

    for( uint i = 1; i <= distance; i++ ){
        uint neighborIndex = this->transCoordToIndex(x + i * dx, y + i * dy, z + i * dz);

        this->setFieldEntry(neighborIndex, type2);
    }
}

HOSTDEVICE void GridImp::findStopperNode(uint index) // deprecated
{
    if(isValidEndOfGridStopper(index))
        this->field.setFieldEntryToStopperOutOfGrid(index);

    if (isValidSolidStopper(index))
        this->field.setFieldEntry(index, STOPPER_SOLID);
}

HOSTDEVICE void GridImp::findEndOfGridStopperNode(uint index)
{
	if (isValidEndOfGridStopper(index)){
        if( this->level != 0 )
		    this->field.setFieldEntryToStopperOutOfGrid(index);
        else
            this->field.setFieldEntryToStopperOutOfGridBoundary(index);
    }

	if (isValidEndOfGridBoundaryStopper(index))
		this->field.setFieldEntryToStopperOutOfGridBoundary(index);
}

HOSTDEVICE void GridImp::findSolidStopperNode(uint index)
{
	if (isValidSolidStopper(index))
		this->field.setFieldEntry(index, STOPPER_SOLID);
}

HOSTDEVICE void GridImp::findBoundarySolidNode(uint index)
{
	if (shouldBeBoundarySolidNode(index)) 
	{
		this->field.setFieldEntry(index, BC_SOLID);
		this->qIndices[index] = this->numberOfSolidBoundaryNodes++;
		//grid->setNumberOfSolidBoundaryNodes(grid->getNumberOfSolidBoundaryNodes() + 1);
	}
}

HOSTDEVICE void GridImp::fixOddCell(uint index)
{
    Cell cell = getOddCellFromIndex(index);
    if (isOutSideOfGrid(cell))
        return;
    if (contains(cell, FLUID))
        setNodeTo(cell, FLUID);
}

HOSTDEVICE bool GridImp::isOutSideOfGrid(Cell &cell) const
{
    for (const auto point : cell) {
        if (point.x < startX || point.x > endX
            || point.y < startY || point.y > endY
            || point.z < startZ || point.z > endZ)
            return true;
    }
    return false;
}

HOSTDEVICE bool GridImp::contains(Cell &cell, char type) const
{
    for (const auto point : cell) {
		uint index = transCoordToIndex(point.x, point.y, point.z);
		if (index == INVALID_INDEX)
			continue;
        if (field.is(index, type))
            return true;
    }
    return false;
}

HOSTDEVICE bool GridImp::cellContainsOnly(Cell &cell, char type) const
{
    for (const auto point : cell) {
		uint index = transCoordToIndex(point.x, point.y, point.z);
		if (index == INVALID_INDEX)
            return false;
        if (!field.is(index, type))
            return false;
    }
    return true;
}

HOSTDEVICE bool GridImp::cellContainsOnly(Cell &cell, char typeA, char typeB) const
{
    for (const auto point : cell) {
		uint index = transCoordToIndex(point.x, point.y, point.z);
		if (index == INVALID_INDEX)
            return false;
        if (!field.is(index, typeA) && !field.is(index, typeB))
            return false;
    }
    return true;
}

HOSTDEVICE const Object * GridImp::getObject() const
{
    return this->object;
}

HOSTDEVICE void GridImp::setNodeTo(Cell &cell, char type)
{
    for (const auto point : cell) {
		uint index = transCoordToIndex(point.x, point.y, point.z);
		if (index == INVALID_INDEX)
			continue;
		field.setFieldEntry(index, type);
    }
}

HOSTDEVICE void GridImp::setNodeTo(uint index, char type)
{
	if( index != INVALID_INDEX )
		field.setFieldEntry(index, type);
}

HOSTDEVICE bool GridImp::isNode(uint index, char type) const
{
    if( index != INVALID_INDEX )
		return field.is(index, type);
}

HOSTDEVICE bool GridImp::isValidEndOfGridStopper(uint index) const
{
	// Lenz: also includes corner stopper nodes
	if (!this->field.is(index, INVALID_OUT_OF_GRID))
		return false;

	return hasNeighborOfType(index, FLUID);

	//previous version of S�ren P.
    //return this->field.is(index, OUT_OF_GRID) && (nodeInNextCellIs(index, FLUID) || nodeInNextCellIs(index, FLUID_CFF))
    //    || this->field.is(index, OUT_OF_GRID) && (nodeInPreviousCellIs(index, FLUID) || nodeInPreviousCellIs(index, FLUID_CFF));
}

HOSTDEVICE bool GridImp::isValidEndOfGridBoundaryStopper(uint index) const
{
	// Lenz: also includes corner stopper nodes
	if (!this->field.is(index, FLUID))
		return false;

	return ! hasAllNeighbors(index);

	//previous version of S�ren P.
    //return this->field.is(index, OUT_OF_GRID) && (nodeInNextCellIs(index, FLUID) || nodeInNextCellIs(index, FLUID_CFF))
    //    || this->field.is(index, OUT_OF_GRID) && (nodeInPreviousCellIs(index, FLUID) || nodeInPreviousCellIs(index, FLUID_CFF));
}

HOSTDEVICE bool GridImp::isValidSolidStopper(uint index) const
{
	// Lenz: also includes corner stopper nodes
	if (!this->field.is(index, INVALID_SOLID))
		return false;

	return hasNeighborOfType(index, FLUID);

	//previous version of S�ren P.
	//return this->field.is(index, SOLID) && (nodeInNextCellIs(index, FLUID) || nodeInNextCellIs(index, FLUID_CFF))
 //       || this->field.is(index, SOLID) && (nodeInPreviousCellIs(index, FLUID) || nodeInPreviousCellIs(index, FLUID_CFF));
}

HOSTDEVICE bool GridImp::shouldBeBoundarySolidNode(uint index) const
{
	if (!this->field.is(index, FLUID))
		return false;

	return hasNeighborOfType(index, STOPPER_SOLID);
}

HOSTDEVICE bool GridImp::hasAllNeighbors(uint index) const
{
	// new version by Lenz, utilizes the range based for loop for all directions
	real x, y, z;
	this->transIndexToCoords(index, x, y, z);
	for (const auto dir : this->distribution) {
		const uint neighborIndex = this->transCoordToIndex(x + dir[0] * this->getDelta(), y + dir[1] * this->getDelta(), z + dir[2] * this->getDelta());

		if (neighborIndex == INVALID_INDEX) return false;
	}

	return true;
}

HOSTDEVICE bool GridImp::hasNeighborOfType(uint index, char type) const
{
	// new version by Lenz, utilizes the range based for loop for all directions
	real x, y, z;
	this->transIndexToCoords(index, x, y, z);
	for (const auto dir : this->distribution) {
		const uint neighborIndex = this->transCoordToIndex(x + dir[0] * this->getDelta(), y + dir[1] * this->getDelta(), z + dir[2] * this->getDelta());

		if (neighborIndex == INVALID_INDEX) continue;

		if (this->field.is(neighborIndex, type))
			return true;
	}

	return false;

    //real x, y, z;
    //this->transIndexToCoords(index, x, y, z);

    //const real neighborX = x + this->delta > endX ? endX : x + this->delta;
    //const real neighborY = y + this->delta > endY ? endY : y + this->delta;
    //const real neighborZ = z + this->delta > endZ ? endZ : z + this->delta;

    //const real neighborMinusX = x - this->delta < startX ? startX : x - this->delta;
    //const real neighborMinusY = y - this->delta < startY ? startY : y - this->delta;
    //const real neighborMinusZ = z - this->delta < startZ ? startZ : z - this->delta;


    //const uint indexMXY = transCoordToIndex(-neighborX, neighborY, z);
    //const uint indexMYZ = transCoordToIndex(x, -neighborY, neighborZ);
    //const uint indexMXZ = transCoordToIndex(-neighborX, y, neighborZ);

    //const uint indexXMY = transCoordToIndex(neighborX, -neighborY, z);
    //const uint indexYMZ = transCoordToIndex(x, neighborY, -neighborZ);
    //const uint indexXMZ = transCoordToIndex(neighborX, y, -neighborZ);

    //const uint indexMXYMZ = transCoordToIndex(-neighborX, neighborY, -neighborZ);
    //const uint indexMXYZ  = transCoordToIndex(-neighborX, neighborY, neighborZ);
    //const uint indexMXMYZ = transCoordToIndex(-neighborX, -neighborY, neighborZ);
    //const uint indexXMYMZ = transCoordToIndex(neighborX, -neighborY, -neighborZ);
    //const uint indexXMYZ  = transCoordToIndex(neighborX, -neighborY, neighborZ);
    //const uint indexXYMZ  = transCoordToIndex(neighborX, neighborY, -neighborZ);



    //return nodeInNextCellIs(index, type) || nodeInPreviousCellIs(index, type) || indexMXY || indexMYZ || indexMXZ || indexXMY || indexYMZ || indexXMZ 
    //    || indexMXYMZ
    //    || indexMXYZ
    //    || indexMXMYZ
    //    || indexXMYMZ
    //    || indexXMYZ
    //    || indexXYMZ;

}

HOSTDEVICE bool GridImp::nodeInNextCellIs(int index, char type) const
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    const real neighborX = x + this->delta > endX ? endX : x + this->delta;
    const real neighborY = y + this->delta > endY ? endY : y + this->delta;
    const real neighborZ = z + this->delta > endZ ? endZ : z + this->delta;

    const uint indexX = transCoordToIndex(neighborX, y, z);
    const uint indexY = transCoordToIndex(x, neighborY, z);
    const uint indexZ = transCoordToIndex(x, y, neighborZ);

    const uint indexXY = transCoordToIndex(neighborX, neighborY, z);
    const uint indexYZ = transCoordToIndex(x, neighborY, neighborZ);
    const uint indexXZ = transCoordToIndex(neighborX, y, neighborZ);

    const uint indexXYZ = transCoordToIndex(neighborX, neighborY, neighborZ);

	const bool typeX   = indexX   == INVALID_INDEX ? false : this->field.is(indexX, type);
	const bool typeY   = indexY   == INVALID_INDEX ? false : this->field.is(indexY, type);
	const bool typeXY  = indexXY  == INVALID_INDEX ? false : this->field.is(indexXY, type);
	const bool typeZ   = indexZ   == INVALID_INDEX ? false : this->field.is(indexZ, type);
	const bool typeYZ  = indexYZ  == INVALID_INDEX ? false : this->field.is(indexYZ, type);
	const bool typeXZ  = indexXZ  == INVALID_INDEX ? false : this->field.is(indexXZ, type);
	const bool typeXYZ = indexXYZ == INVALID_INDEX ? false : this->field.is(indexXYZ, type);

    return typeX || typeY || typeXY || typeZ || typeYZ
        || typeXZ || typeXYZ;
}

HOSTDEVICE bool GridImp::nodeInPreviousCellIs(int index, char type) const
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    const real neighborX = x - this->delta < startX ? startX : x - this->delta;
    const real neighborY = y - this->delta < startY ? startY : y - this->delta;
    const real neighborZ = z - this->delta < startZ ? startZ : z - this->delta;

    const uint indexX = transCoordToIndex(neighborX, y, z);
    const uint indexY = transCoordToIndex(x, neighborY, z);
    const uint indexZ = transCoordToIndex(x, y, neighborZ);

    const uint indexXY = transCoordToIndex(neighborX, neighborY, z);
    const uint indexYZ = transCoordToIndex(x, neighborY, neighborZ);
    const uint indexXZ = transCoordToIndex(neighborX, y, neighborZ);

    const uint indexXYZ = transCoordToIndex(neighborX, neighborY, neighborZ);

	const bool typeX   = indexX   == INVALID_INDEX ? false : this->field.is(indexX  , type);
	const bool typeY   = indexY   == INVALID_INDEX ? false : this->field.is(indexY  , type);
	const bool typeXY  = indexXY  == INVALID_INDEX ? false : this->field.is(indexXY , type);
	const bool typeZ   = indexZ   == INVALID_INDEX ? false : this->field.is(indexZ  , type);
	const bool typeYZ  = indexYZ  == INVALID_INDEX ? false : this->field.is(indexYZ , type);
	const bool typeXZ  = indexXZ  == INVALID_INDEX ? false : this->field.is(indexXZ , type);
	const bool typeXYZ = indexXYZ == INVALID_INDEX ? false : this->field.is(indexXYZ, type);

    return typeX || typeY || typeXY || typeZ || typeYZ
        || typeXZ || typeXYZ;
}

HOSTDEVICE bool GridImp::nodeInCellIs(Cell& cell, char type) const
{
    for (const auto node : cell)
    {
        const uint index = transCoordToIndex(node.x, node.y, node.z);
		if (index == INVALID_INDEX)
			continue;
        if (field.is(index, type))
            return true;
    }
    return false;
}


void GridImp::setCellTo(uint index, char type)
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    Cell cell(x, y, z, this->delta);
    for (const auto node : cell)
    {
        const uint nodeIndex = transCoordToIndex(node.x, node.y, node.z);
		if (nodeIndex == INVALID_INDEX)
			continue;
		this->field.setFieldEntry(nodeIndex, type);
    }
}


void GridImp::setNonStopperOutOfGridCellTo(uint index, char type)
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    Cell cell(x, y, z, this->delta);
    for (const auto node : cell)
    {
        const uint nodeIndex = transCoordToIndex(node.x, node.y, node.z);
		if (nodeIndex == INVALID_INDEX)
			continue;

        if( this->getFieldEntry( nodeIndex ) != STOPPER_OUT_OF_GRID && 
            this->getFieldEntry( nodeIndex ) != STOPPER_OUT_OF_GRID_BOUNDARY )
            this->field.setFieldEntry(nodeIndex, type);
    }
}


HOST void GridImp::setPeriodicity(bool periodicityX, bool periodicityY, bool periodicityZ)
{
    this->periodicityX = periodicityX;
    this->periodicityY = periodicityY;
    this->periodicityZ = periodicityZ;
}

void GridImp::setPeriodicityX(bool periodicity)
{
    this->periodicityX = periodicityX;
}

void GridImp::setPeriodicityY(bool periodicity)
{
    this->periodicityY = periodicityY;
}

void GridImp::setPeriodicityZ(bool periodicity)
{
    this->periodicityZ = periodicityZ;
}

bool GridImp::getPeriodicityX()
{
    return this->periodicityX;
}

bool GridImp::getPeriodicityY()
{
    return this->periodicityY;
}

bool GridImp::getPeriodicityZ()
{
    return this->periodicityZ;
}

void GridImp::setEnableFixRefinementIntoTheWall(bool enableFixRefinementIntoTheWall)
{
    this->enableFixRefinementIntoTheWall = enableFixRefinementIntoTheWall;
}

HOSTDEVICE uint GridImp::transCoordToIndex(const real &x, const real &y, const real &z) const
{
    const uint xIndex = getXIndex(x);
    const uint yIndex = getYIndex(y);
    const uint zIndex = getZIndex(z);

	if (xIndex >= nx || yIndex >= ny || zIndex >= nz)
        return INVALID_INDEX;

    return xIndex + nx * (yIndex + ny * zIndex);
}

HOSTDEVICE void GridImp::transIndexToCoords(uint index, real &x, real &y, real &z) const
{
    if (index == INVALID_INDEX)
        printf("Function: transIndexToCoords. GridImp Index: %d, size: %d. Exit Program!\n", index, size);

    x = index % nx;
    y = (index / nx) % ny;
    z = ((index / nx) / ny) % nz;

    x = (x * delta) + startX;
    y = (y * delta) + startY;
    z = (z * delta) + startZ;
}

HOST uint GridImp::getLevel(real startDelta) const
{
    uint level = 0;
    real delta = this->delta;
    while(!vf::Math::equal(delta, startDelta))
    {
        delta *= 2;
        level++;
    }
    return level;
}

HOST uint GridImp::getLevel() const
{
    return this->level;
}

HOST void GridImp::setTriangularMeshDiscretizationStrategy(TriangularMeshDiscretizationStrategy* triangularMeshDiscretizationStrategy)
{
    this->triangularMeshDiscretizationStrategy = triangularMeshDiscretizationStrategy;
}

HOST TriangularMeshDiscretizationStrategy * GridImp::getTriangularMeshDiscretizationStrategy()
{
    return this->triangularMeshDiscretizationStrategy;
}

HOST uint GridImp::getNumberOfSolidBoundaryNodes() const
{
	return this->numberOfSolidBoundaryNodes;
}

HOST void GridImp::setNumberOfSolidBoundaryNodes(uint numberOfSolidBoundaryNodes)
{
	if (numberOfSolidBoundaryNodes >= 0 && numberOfSolidBoundaryNodes < INVALID_INDEX)
		this->numberOfSolidBoundaryNodes = numberOfSolidBoundaryNodes;
}

HOST real GridImp::getQValue(const uint index, const uint dir) const
{
	const int qIndex = dir * this->numberOfSolidBoundaryNodes + this->qIndices[index];

	return this->qValues[qIndex];
}

HOST uint GridImp::getQPatch(const uint index) const
{
    return this->qPatches[ this->qIndices[index] ];
}

HOST void GridImp::setInnerRegionFromFinerGrid(bool innerRegionFromFinerGrid)
{
   this->innerRegionFromFinerGrid = innerRegionFromFinerGrid;
}

HOST void GridImp::setNumberOfLayers(uint numberOfLayers)
{
    this->numberOfLayers = numberOfLayers;
}

// --------------------------------------------------------- //
//                  Set Sparse Indices                       //
// --------------------------------------------------------- //

HOST void GridImp::findSparseIndices(SPtr<Grid> fineGrid)
{
    this->gridStrategy->findSparseIndices(shared_from_this(), std::static_pointer_cast<GridImp>(fineGrid));
}


HOST void GridImp::updateSparseIndices()
{
    int removedNodes = 0;
    int newIndex = 0;
    for (uint index = 0; index < size; index++)
    {
        if (this->field.isInvalidCoarseUnderFine(index) || this->field.isInvalidOutOfGrid(index) || this->field.isInvalidSolid(index))
        {
            sparseIndices[index] = -1;
            removedNodes++;
        }
        else
        {
            sparseIndices[index] = newIndex;
            newIndex++;
        }
    }
    sparseSize = size - removedNodes;
}

HOSTDEVICE void GridImp::setNeighborIndices(uint index)
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    this->neighborIndexX[index]        = -1;
    this->neighborIndexY[index]        = -1;
    this->neighborIndexZ[index]        = -1;
    this->neighborIndexNegative[index] = -1;

    if (this->field.isStopper(index) || this->field.is(index, STOPPER_OUT_OF_GRID_BOUNDARY))
    {
        this->setStopperNeighborCoords(index);
        return;
    }
     
    if (this->sparseIndices[index] == -1)
        return;


    real neighborXCoord, neighborYCoord, neighborZCoord;
    this->getNeighborCoords(neighborXCoord, neighborYCoord, neighborZCoord, x, y, z);
    const int neighborX = getSparseIndex(neighborXCoord, y, z);
    const int neighborY = getSparseIndex(x, neighborYCoord, z);
    const int neighborZ = getSparseIndex(x, y, neighborZCoord);

    this->getNegativeNeighborCoords(neighborXCoord, neighborYCoord, neighborZCoord, x, y, z);
    const int neighborNegative = getSparseIndex(neighborXCoord, neighborYCoord, neighborZCoord);

    this->neighborIndexX[index]        = neighborX;
    this->neighborIndexY[index]        = neighborY;
    this->neighborIndexZ[index]        = neighborZ;
    this->neighborIndexNegative[index] = neighborNegative;
}

HOSTDEVICE void GridImp::setStopperNeighborCoords(uint index)
{
    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    if (vf::Math::lessEqual(x + delta, endX) && !this->field.isInvalidOutOfGrid(this->transCoordToIndex(x + delta, y, z)))
        neighborIndexX[index] = getSparseIndex(x + delta, y, z);

    if (vf::Math::lessEqual(y + delta, endY) && !this->field.isInvalidOutOfGrid(this->transCoordToIndex(x, y + delta, z)))
        neighborIndexY[index] = getSparseIndex(x, y + delta, z);

    if (vf::Math::lessEqual(z + delta, endZ) && !this->field.isInvalidOutOfGrid(this->transCoordToIndex(x, y, z + delta)))
        neighborIndexZ[index] = getSparseIndex(x, y, z + delta);

    if (vf::Math::greaterEqual(x - delta, endX) && 
        vf::Math::greaterEqual(y - delta, endY) && 
        vf::Math::greaterEqual(z - delta, endZ) && 
        !this->field.isInvalidOutOfGrid(this->transCoordToIndex(x - delta, y - delta, z - delta)))
    {
        neighborIndexNegative[index] = getSparseIndex(x - delta, y - delta, z - delta);
    }
}

HOSTDEVICE void GridImp::getNeighborCoords(real &neighborX, real &neighborY, real &neighborZ, real x, real y, real z) const
{
    real coords[3] = { x, y, z };
    neighborX = getNeighborCoord(periodicityX, startX, coords, 0);
    neighborY = getNeighborCoord(periodicityY, startY, coords, 1);
    neighborZ = getNeighborCoord(periodicityZ, startZ, coords, 2);
}

HOSTDEVICE real GridImp::getNeighborCoord(bool periodicity, real startCoord, real coords[3], int direction) const
{
    if (periodicity)
    {
        real neighborCoords[3] = {coords[0], coords[1] , coords[2] };
        neighborCoords[direction] = neighborCoords[direction] + delta;
        const int neighborIndex = this->transCoordToIndex(neighborCoords[0], neighborCoords[1], neighborCoords[2]);

        //if(!field.isStopperOutOfGrid(neighborIndex) && !field.is(neighborIndex, STOPPER_OUT_OF_GRID_BOUNDARY) )
        //    return coords[direction] + delta;

        //return getFirstFluidNode(coords, direction, startCoord);

        //////////////////////////////////////////////////////////////////////////

        if( field.is(neighborIndex, STOPPER_OUT_OF_GRID_BOUNDARY) )
            return getFirstFluidNode(coords, direction, startCoord);
        else
            return coords[direction] + delta;

    }
    
    return coords[direction] + delta;
}

HOSTDEVICE void GridImp::getNegativeNeighborCoords(real &neighborX, real &neighborY, real &neighborZ, real x, real y, real z) const
{
    real coords[3] = { x, y, z };
    neighborX = getNegativeNeighborCoord(periodicityX, startX, coords, 0);
    neighborY = getNegativeNeighborCoord(periodicityY, startY, coords, 1);
    neighborZ = getNegativeNeighborCoord(periodicityZ, startZ, coords, 2);
}

HOSTDEVICE real GridImp::getNegativeNeighborCoord(bool periodicity, real startCoord, real coords[3], int direction) const
{
    if (periodicity)
    {
        real neighborCoords[3] = {coords[0], coords[1] , coords[2] };
        neighborCoords[direction] = neighborCoords[direction] - delta;
        const int neighborIndex = this->transCoordToIndex(neighborCoords[0], neighborCoords[1], neighborCoords[2]);

        if(neighborIndex != INVALID_INDEX && !field.isStopperOutOfGrid(neighborIndex) && !field.is(neighborIndex, STOPPER_OUT_OF_GRID_BOUNDARY) )
            return coords[direction] - delta;

        return getLastFluidNode(coords, direction, startCoord);
    }
    
    return coords[direction] - delta;
}


HOSTDEVICE real GridImp::getLastFluidNode(real coords[3], int direction, real startCoord) const
{
    coords[direction] = startCoord;
    int index = this->transCoordToIndex(coords[0], coords[1], coords[2]);
    while (index != INVALID_INDEX && !field.isFluid(index))
    {
        coords[direction] -= delta;
        index = this->transCoordToIndex(coords[0], coords[1], coords[2]);
    }
    return coords[direction];
}

HOSTDEVICE real GridImp::getFirstFluidNode(real coords[3], int direction, real startCoord) const
{
    coords[direction] = startCoord;
    uint index = this->transCoordToIndex(coords[0], coords[1], coords[2]);
    while (!field.isFluid(index))
    {
        coords[direction] += delta;
        index = this->transCoordToIndex(coords[0], coords[1], coords[2]);
    }
    return coords[direction];
}


HOSTDEVICE int GridImp::getSparseIndex(const real &x, const real &y, const real &z) const
{
    const int matrixIndex = transCoordToIndex(x, y, z);
    return sparseIndices[matrixIndex];
}


// --------------------------------------------------------- //
//                    Find Interface                         //
// --------------------------------------------------------- //
HOST void GridImp::findGridInterface(SPtr<Grid> finerGrid, LbmOrGks lbmOrGks)
{
    gridStrategy->findGridInterface(shared_from_this(), std::static_pointer_cast<GridImp>(finerGrid), lbmOrGks);
}

HOSTDEVICE void GridImp::repairGridInterfaceOnMultiGPU(SPtr<Grid> fineGrid)
{
    this->gridInterface->repairGridInterfaceOnMultiGPU( shared_from_this(), std::static_pointer_cast<GridImp>(fineGrid) );
}

HOST void GridImp::limitToSubDomain(SPtr<BoundingBox> subDomainBox)
{
    for( uint index = 0; index < this->size; index++ ){

        real x, y, z;
        this->transIndexToCoords( index, x, y, z );

        {
            BoundingBox tmpSubDomainBox = *subDomainBox;

            // one layer for receive nodes and one for stoppers
            tmpSubDomainBox.extend(this->delta);

            if (!tmpSubDomainBox.isInside(x, y, z))
                this->setFieldEntry(index, STOPPER_OUT_OF_GRID_BOUNDARY);
        }

        {
            BoundingBox tmpSubDomainBox = *subDomainBox;

            // one layer for receive nodes and one for stoppers
            tmpSubDomainBox.extend(2.0 * this->delta);

            if (!tmpSubDomainBox.isInside(x, y, z))
                this->setFieldEntry(index, INVALID_OUT_OF_GRID);
        }
    }

    //this->gridStrategy->findEndOfGridStopperNodes(shared_from_this());
}

HOSTDEVICE void GridImp::findGridInterfaceCF(uint index, GridImp& finerGrid, LbmOrGks lbmOrGks)
{
	if (lbmOrGks == LBM)
	{
		gridInterface->findInterfaceCF            (index, this, &finerGrid);
		gridInterface->findBoundaryGridInterfaceCF(index, this, &finerGrid);
	}
	else if (lbmOrGks == GKS)
		gridInterface->findInterfaceCF_GKS(index, this, &finerGrid);
}

HOSTDEVICE void GridImp::findGridInterfaceFC(uint index, GridImp& finerGrid)
{
    gridInterface->findInterfaceFC(index, this, &finerGrid);
}

HOSTDEVICE void GridImp::findOverlapStopper(uint index, GridImp& finerGrid)
{
    gridInterface->findOverlapStopper(index, this, &finerGrid);
}

// --------------------------------------------------------- //
//                    Mesh Triangle                          //
// --------------------------------------------------------- //
HOST void GridImp::mesh(Object* object)
{
    TriangularMesh* triangularMesh = dynamic_cast<TriangularMesh*>(object);
    if (triangularMesh)
        triangularMeshDiscretizationStrategy->discretize(triangularMesh, this, INVALID_SOLID, FLUID);
    else
        gridStrategy->findInnerNodes(shared_from_this()); //TODO: adds INNERTYPE AND OUTERTYPE to findInnerNodes 
		//new method for geometric primitives (not cell based) to be implemented

    this->closeNeedleCells();

	//gridStrategy->findStopperNodes(shared_from_this()); //deprecated
	gridStrategy->findSolidStopperNodes(shared_from_this());
	gridStrategy->findBoundarySolidNodes(shared_from_this());
}


HOST void GridImp::mesh(TriangularMesh &triangularMesh)
{
    const clock_t begin = clock();

    gridStrategy->mesh(shared_from_this(), triangularMesh);

    const clock_t end = clock();
    const real time = (real)(real(end - begin) / CLOCKS_PER_SEC);

    *logging::out << logging::Logger::INFO_INTERMEDIATE << "time grid generation: " << time << "s\n";
}

HOSTDEVICE void GridImp::mesh(Triangle &triangle)
{
    auto box = this->getBoundingBoxOnNodes(triangle);
    triangle.initalLayerThickness(getDelta());

    for (real x = box.minX; x <= box.maxX; x += delta)
    {
        for (real y = box.minY; y <= box.maxY; y += delta)
        {
            for (real z = box.minZ; z <= box.maxZ; z += delta)
            {
                const uint index = this->transCoordToIndex(x, y, z);
                if (!field.isFluid(index))
                    continue;

                const Vertex point(x, y, z);
                const int value = triangle.isUnderFace(point);
                //setDebugPoint(index, value);

                if (value == Q_DEPRECATED)
                    calculateQs(point, triangle);
            }
        }
    }
}

HOST void GridImp::closeNeedleCells()
{
    *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start closeNeedleCells()\n";

    uint numberOfClosedNeedleCells = 0;

    do{
        numberOfClosedNeedleCells = this->gridStrategy->closeNeedleCells( shared_from_this() );
        *logging::out << logging::Logger::INFO_INTERMEDIATE << numberOfClosedNeedleCells << " cells closed!\n";
    }
    while( numberOfClosedNeedleCells > 0 );
}

HOSTDEVICE bool GridImp::closeCellIfNeedle(uint index)
{
    if( !this->getField().is( index, FLUID ) ) return false;

    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    bool noValidNeighborInX = this->getField().is( this->transCoordToIndex( x + this->delta, y,               z               ) , INVALID_SOLID ) &&
                              this->getField().is( this->transCoordToIndex( x - this->delta, y,               z               ) , INVALID_SOLID );
    bool noValidNeighborInY = this->getField().is( this->transCoordToIndex( x,               y + this->delta, z               ) , INVALID_SOLID ) &&
                              this->getField().is( this->transCoordToIndex( x,               y - this->delta, z               ) , INVALID_SOLID );
    bool noValidNeighborInZ = this->getField().is( this->transCoordToIndex( x,               y,               z + this->delta ) , INVALID_SOLID ) &&
                              this->getField().is( this->transCoordToIndex( x,               y,               z - this->delta ) , INVALID_SOLID );

    if( noValidNeighborInX || noValidNeighborInY || noValidNeighborInZ ){
        this->setFieldEntry(index, INVALID_SOLID);
        return true;
    }

    return false;
}

HOST void GridImp::closeNeedleCellsThinWall()
{
    *logging::out << logging::Logger::INFO_INTERMEDIATE << "Start closeNeedleCellsThinWall()\n";

    uint numberOfClosedNeedleCells = 0;

    do{
        numberOfClosedNeedleCells = this->gridStrategy->closeNeedleCellsThinWall( shared_from_this() );
        *logging::out << logging::Logger::INFO_INTERMEDIATE << numberOfClosedNeedleCells << " cells closed!\n";
    }
    while( numberOfClosedNeedleCells > 0 );
}

HOSTDEVICE bool GridImp::closeCellIfNeedleThinWall(uint index)
{
    if( !this->getField().is( index, BC_SOLID ) ) return false;

    real x, y, z;
    this->transIndexToCoords(index, x, y, z);

    //bool noValidNeighborInX = !this->getField().is( this->transCoordToIndex( x + this->delta, y,               z               ) , FLUID ) &&
    //                          !this->getField().is( this->transCoordToIndex( x - this->delta, y,               z               ) , FLUID );
    //bool noValidNeighborInY = !this->getField().is( this->transCoordToIndex( x,               y + this->delta, z               ) , FLUID ) &&
    //                          !this->getField().is( this->transCoordToIndex( x,               y - this->delta, z               ) , FLUID );
    //bool noValidNeighborInZ = !this->getField().is( this->transCoordToIndex( x,               y,               z + this->delta ) , FLUID ) &&
    //                          !this->getField().is( this->transCoordToIndex( x,               y,               z - this->delta ) , FLUID );

    //if( noValidNeighborInX && noValidNeighborInY && noValidNeighborInZ ){
    //    this->setFieldEntry(index, STOPPER_SOLID);
    //    return true;
    //}

    if( !this->hasNeighborOfType(index, FLUID) ){
        this->setFieldEntry(index, STOPPER_SOLID);
        return true;
    }

    return false;
}



HOST void GridImp::findQs(Object* object) //TODO: enable qs for primitive objects
{
    TriangularMesh* triangularMesh = dynamic_cast<TriangularMesh*>(object);
    if (triangularMesh)
        findQs(*triangularMesh);
}

HOST void GridImp::findQs(TriangularMesh &triangularMesh)
{
    const clock_t begin = clock();

    if( this->qComputationStage == qComputationStageType::ComputeQs ){
	    gridStrategy->allocateQs(shared_from_this());
    }

    gridStrategy->findQs(shared_from_this(), triangularMesh);

    const clock_t end = clock();
    const real time = (real)(real(end - begin) / CLOCKS_PER_SEC);

    *logging::out << logging::Logger::INFO_INTERMEDIATE << "time finding qs: " << time << "s\n";
}

HOSTDEVICE void GridImp::findQs(Triangle &triangle)
{
    auto box = this->getBoundingBoxOnNodes(triangle);
    triangle.initalLayerThickness(getDelta());

    for (real x = box.minX; x <= box.maxX; x += delta)
    {
        for (real y = box.minY; y <= box.maxY; y += delta)
        {
            for (real z = box.minZ; z <= box.maxZ; z += delta)
            {
                const uint index = this->transCoordToIndex(x, y, z);
                //if (!field.isFluid(index))
                //    continue;

				if( index == INVALID_INDEX ) continue;

                const Vertex point(x, y, z);

                if( this->qComputationStage == qComputationStageType::ComputeQs ){
                    if(this->field.is(index, BC_SOLID))
                    {
					    calculateQs(index, point, triangle);
				    }
                }
                else if( this->qComputationStage == qComputationStageType::FindSolidBoundaryNodes )
                {
                    //if( this->field.is(index, BC_SOLID) || this->field.is(index, STOPPER_SOLID ) ) continue;

                    if( !this->field.is(index, FLUID) ) continue;

                    if( checkIfAtLeastOneValidQ(index, point, triangle) )
                    {
                        // similar as in void GridImp::findBoundarySolidNode(uint index)
                        this->field.setFieldEntry( index, BC_SOLID );
                        this->qIndices[index] = this->numberOfSolidBoundaryNodes++;
                    }
                }
            }
        }
    }
}

HOSTDEVICE void GridImp::setDebugPoint(uint index, int pointValue)
{
    if (field.isInvalidCoarseUnderFine(index) && pointValue == INVALID_SOLID)
        field.setFieldEntry(index, pointValue);

    if(!field.isInvalidSolid(index) && !field.isQ(index) && !field.isInvalidCoarseUnderFine(index) && pointValue != 3 && pointValue != 2)
        field.setFieldEntry(index, pointValue);
}

HOSTDEVICE void GridImp::calculateQs(const Vertex &point, const Triangle &triangle) const   // NOT USED !!!!
{
    Vertex pointOnTriangle, direction;
    //VertexInteger solid_node;
    real subdistance;
    int error;
    for (int i = distribution.dir_start; i <= distribution.dir_end; i++)
    {
#if defined(__CUDA_ARCH__)
        direction = Vertex(DIRECTIONS[i][0], DIRECTIONS[i][1], DIRECTIONS[i][2]);
#else
        direction = Vertex(real(distribution.dirs[i * DIMENSION + 0]), real(distribution.dirs[i * DIMENSION + 1]),
            real(distribution.dirs[i * DIMENSION + 2]));
#endif

        error = triangle.getTriangleIntersection(point, direction, pointOnTriangle, subdistance);

		//real lengthDirection = sqrt(direction.x * direction.x + direction.y * direction.y + direction.z * direction.z);
		subdistance /= /*lengthDirection **/ this->delta;

        if (error == 0 && subdistance < 1.0 && subdistance > 0.0)
        {
            //solid_node = VertexInteger(actualPoint.x + direction.x, actualPoint.y + direction.y, actualPoint.z + direction.z);
            distribution.f[i*size + transCoordToIndex(point.x, point.y, point.z)] = subdistance;
            //printf("Q%d %d: %2.8f \n", i, grid.transCoordToIndex(actualPoint), grid.d.f[index]);
        } /*else
            distribution.f[i*size + transCoordToIndex(point.x, point.y, point.z)] = -1.0;*/
    }
}


HOSTDEVICE void GridImp::calculateQs(const uint index, const Vertex &point, const Triangle &triangle) const
{
	Vertex pointOnTriangle, direction;
	//VertexInteger solid_node;
	real subdistance;
	int error;
	for (int i = distribution.dir_start; i <= distribution.dir_end; i++)
	{
#if defined(__CUDA_ARCH__)
		direction = Vertex(DIRECTIONS[i][0], DIRECTIONS[i][1], DIRECTIONS[i][2]);
#else
		direction = Vertex( real(distribution.dirs[i * DIMENSION + 0]), 
                            real(distribution.dirs[i * DIMENSION + 1]),
			                real(distribution.dirs[i * DIMENSION + 2]) );
#endif

		uint neighborIndex = this->transCoordToIndex(point.x + direction.x * this->delta,
													 point.y + direction.y * this->delta,
													 point.z + direction.z * this->delta);

		if (neighborIndex == INVALID_INDEX) continue;

		//if ( this->field.is(neighborIndex, STOPPER_SOLID) )
  //      {
  //          if (this->qValues[i*this->numberOfSolidBoundaryNodes + this->qIndices[index]] < -0.5)
  //          {
  //              this->qValues[i*this->numberOfSolidBoundaryNodes + this->qIndices[index]] = 1.0;
  //          }
  //      }

		error = triangle.getTriangleIntersection(point, direction, pointOnTriangle, subdistance);

		subdistance /= this->delta;

		if (error == 0 && vf::Math::lessEqual(subdistance, 1.0) && vf::Math::greaterEqual(subdistance, 0.0))
		{
			if ( -0.5        > this->qValues[i*this->numberOfSolidBoundaryNodes + this->qIndices[index]] ||
                 subdistance < this->qValues[i*this->numberOfSolidBoundaryNodes + this->qIndices[index]] )
			{
				this->qValues[i*this->numberOfSolidBoundaryNodes + this->qIndices[index]] = subdistance;

                this->qPatches[ this->qIndices[index] ] = triangle.patchIndex;

				//printf("%d %f \n", this->qIndices[index], subdistance);
			}
		}
	}
}

HOSTDEVICE bool GridImp::checkIfAtLeastOneValidQ(const uint index, const Vertex & point, const Triangle & triangle) const
{
	Vertex pointOnTriangle, direction;
	//VertexInteger solid_node;
	real subdistance;
	int error;
	for (int i = distribution.dir_start; i <= distribution.dir_end; i++)
	{
#if defined(__CUDA_ARCH__)
		direction = Vertex(DIRECTIONS[i][0], DIRECTIONS[i][1], DIRECTIONS[i][2]);
#else
		direction = Vertex(real(distribution.dirs[i * DIMENSION + 0]), real(distribution.dirs[i * DIMENSION + 1]),
			real(distribution.dirs[i * DIMENSION + 2]));
#endif

		uint neighborIndex = this->transCoordToIndex(point.x + direction.x * this->delta,
													 point.y + direction.y * this->delta,
													 point.z + direction.z * this->delta);
		if (neighborIndex == INVALID_INDEX) continue;

		error = triangle.getTriangleIntersection(point, direction, pointOnTriangle, subdistance);

		subdistance /= this->delta;

		if (error == 0 && vf::Math::lessEqual(subdistance, 1.0) && vf::Math::greaterEqual(subdistance, 0.0))
		{
			return true;
		}
	}
    return false;
}

void GridImp::findCommunicationIndices(int direction, SPtr<BoundingBox> subDomainBox)
{
    for( uint index = 0; index < this->size; index++ ){
        
        real x, y, z;
        this->transIndexToCoords(index, x, y, z);
    
        if( this->getFieldEntry(index) == INVALID_OUT_OF_GRID ||
            this->getFieldEntry(index) == INVALID_SOLID ||
            this->getFieldEntry(index) == INVALID_COARSE_UNDER_FINE ||
            this->getFieldEntry(index) == STOPPER_SOLID ||
            this->getFieldEntry(index) == STOPPER_OUT_OF_GRID ||
            this->getFieldEntry(index) == STOPPER_OUT_OF_GRID_BOUNDARY ||
            this->getFieldEntry(index) == STOPPER_COARSE_UNDER_FINE ) continue;

        if( direction == CommunicationDirections::MX ) findCommunicationIndex( index, x, subDomainBox->minX, direction);
        if( direction == CommunicationDirections::PX ) findCommunicationIndex( index, x, subDomainBox->maxX, direction);
        if( direction == CommunicationDirections::MY ) findCommunicationIndex( index, y, subDomainBox->minY, direction);
        if( direction == CommunicationDirections::PY ) findCommunicationIndex( index, y, subDomainBox->maxY, direction);
        if( direction == CommunicationDirections::MZ ) findCommunicationIndex( index, z, subDomainBox->minZ, direction);
        if( direction == CommunicationDirections::PZ ) findCommunicationIndex( index, z, subDomainBox->maxZ, direction);
    }
}

void GridImp::findCommunicationIndex( uint index, real coordinate, real limit, int direction ){
        
    // negative direction get a negative sign
    real s = ( direction % 2 == 0 ) ? ( -1.0 ) : ( 1.0 );  


	if (std::abs(coordinate - (limit + s * 0.5 * this->delta)) < 0.01 * this->delta) {
		this->communicationIndices[direction].receiveIndices.push_back(index);
	}

	if ( std::abs( coordinate - ( limit - s * 0.5 * this->delta ) ) < 0.01 * this->delta) {
		this->communicationIndices[direction].sendIndices.push_back(index);
	}
}

uint GridImp::getNumberOfSendNodes(int direction)
{
    return this->communicationIndices[direction].sendIndices.size();
}

uint GridImp::getNumberOfReceiveNodes(int direction)
{
    return this->communicationIndices[direction].receiveIndices.size();
}

uint GridImp::getSendIndex(int direction, uint index)
{
    return this->communicationIndices[direction].sendIndices[ index ];
}

uint GridImp::getReceiveIndex(int direction, uint index)
{
    return this->communicationIndices[direction].receiveIndices[ index ];
}


// --------------------------------------------------------- //
//                        Getter                             //
// --------------------------------------------------------- //
HOSTDEVICE int GridImp::getSparseIndex(uint matrixIndex) const
{
    return this->sparseIndices[matrixIndex];
}

HOST real* GridImp::getDistribution() const
{
    return this->distribution.f;
}

HOST int* GridImp::getDirection() const
{
    return this->distribution.dirs;
}

HOST int GridImp::getStartDirection() const
{
    return this->distribution.dir_start;
}

HOST int GridImp::getEndDirection() const
{
    return this->distribution.dir_end;
}

HOSTDEVICE BoundingBox GridImp::getBoundingBoxOnNodes(Triangle &triangle) const
{
    real minX, maxX, minY, maxY, minZ, maxZ;
    triangle.setMinMax(minX, maxX, minY, maxY, minZ, maxZ);
    //const Vertex minOnNodes = getMinimumOnNode(Vertex(minX, minY, minZ));
    //const Vertex maxOnNodes = getMaximumOnNode(Vertex(maxX, maxY, maxZ));

	int minXIndex = lround(floor((minX - this->startX) / this->delta)) - 1;
	int minYIndex = lround(floor((minY - this->startY) / this->delta)) - 1;
	int minZIndex = lround(floor((minZ - this->startZ) / this->delta)) - 1;

	int maxXIndex = lround(ceil ((maxX - this->startX) / this->delta)) + 1;
	int maxYIndex = lround(ceil ((maxY - this->startY) / this->delta)) + 1;
	int maxZIndex = lround(ceil ((maxZ - this->startZ) / this->delta)) + 1;

	minX = this->startX + minXIndex * this->delta;
	minY = this->startY + minYIndex * this->delta;
	minZ = this->startZ + minZIndex * this->delta;

	maxX = this->startX + maxXIndex * this->delta;
	maxY = this->startY + maxYIndex * this->delta;
	maxZ = this->startZ + maxZIndex * this->delta;

    //return BoundingBox(minOnNodes.x, maxOnNodes.x, minOnNodes.y, maxOnNodes.y, minOnNodes.z, maxOnNodes.z);
	return BoundingBox( minX, maxX, minY, maxY, minZ, maxZ );
}

HOSTDEVICE Vertex GridImp::getMinimumOnNode(Vertex exact) const  //deprecated
{
    const real minX = getMinimumOnNodes(exact.x, vf::Math::getDecimalPart(startX), delta);
    const real minY = getMinimumOnNodes(exact.y, vf::Math::getDecimalPart(startY), delta);
    const real minZ = getMinimumOnNodes(exact.z, vf::Math::getDecimalPart(startZ), delta);
    return Vertex(minX, minY, minZ);
}

HOSTDEVICE real GridImp::getMinimumOnNodes(const real& minExact, const real& decimalStart, const real& delta)  //deprecated
{
    real minNode = ceil(minExact - 1.0);
    minNode += decimalStart;
    while (minNode > minExact)
        minNode -= delta;

    while (minNode + delta < minExact)
        minNode += delta;
    return minNode;
}

HOSTDEVICE Vertex GridImp::getMaximumOnNode(Vertex exact) const  //deprecated
{
    const real maxX = getMaximumOnNodes(exact.x, vf::Math::getDecimalPart(startX), delta);
    const real maxY = getMaximumOnNodes(exact.y, vf::Math::getDecimalPart(startY), delta);
    const real maxZ = getMaximumOnNodes(exact.z, vf::Math::getDecimalPart(startZ), delta);
    return Vertex(maxX, maxY, maxZ);
}

HOSTDEVICE real GridImp::getMaximumOnNodes(const real& maxExact, const real& decimalStart, const real& delta)  //deprecated
{
    real maxNode = ceil(maxExact - 1.0);
    maxNode += decimalStart;

    while (maxNode <= maxExact)
        maxNode += delta;
    return maxNode;
}

HOSTDEVICE uint GridImp::getXIndex(real x) const
{
    return lround((x - startX) / delta);
	//return int((x - startX) / delta);
}

HOSTDEVICE uint GridImp::getYIndex(real y) const
{
    return lround((y - startY) / delta);
	//return int((y - startY) / delta);
}

HOSTDEVICE uint GridImp::getZIndex(real z) const
{
	return lround((z - startZ) / delta);
	//return int((z - startZ) / delta);
}

HOSTDEVICE real GridImp::getDelta() const
{
    return delta;
}

HOSTDEVICE uint GridImp::getSize() const
{
    return this->size;
}

HOSTDEVICE uint GridImp::getSparseSize() const
{
    return this->sparseSize;
}

HOSTDEVICE Field GridImp::getField() const
{
    return this->field;
}

char GridImp::getFieldEntry(uint index) const
{
    return this->field.getFieldEntry(index);
}

HOSTDEVICE void GridImp::setFieldEntry(uint matrixIndex, char type)
{
    this->field.setFieldEntry(matrixIndex, type);
}


real GridImp::getStartX() const
{
    return startX;
}

real GridImp::getStartY() const
{
    return startY;
}

real GridImp::getStartZ() const
{
    return startZ;
}

real GridImp::getEndX() const
{
    return endX;
}

real GridImp::getEndY() const
{
    return endY;
}

real GridImp::getEndZ() const
{
    return endZ;
}

uint GridImp::getNumberOfNodesX() const
{
    return nx;
}

uint GridImp::getNumberOfNodesY() const
{
    return ny;
}

uint GridImp::getNumberOfNodesZ() const
{
    return nz;
}

SPtr<GridStrategy> GridImp::getGridStrategy() const
{
    return gridStrategy;
}


int* GridImp::getNeighborsX() const
{
    return this->neighborIndexX;
}

int* GridImp::getNeighborsY() const
{
    return this->neighborIndexY;
}

int* GridImp::getNeighborsZ() const
{
    return this->neighborIndexZ;
}

int* GridImp::getNeighborsNegative() const
{
    return this->neighborIndexNegative;
}


uint GridImp::getNumberOfNodesCF() const
{
    if(this->gridInterface)
        return this->gridInterface->cf.numberOfEntries;
    return 0;
}

uint GridImp::getNumberOfNodesFC() const
{
    if (this->gridInterface)
        return this->gridInterface->fc.numberOfEntries;
    return 0;
}

uint* GridImp::getCF_coarse() const
{
    return this->gridInterface->cf.coarse;
}

uint* GridImp::getCF_fine() const
{
    return this->gridInterface->cf.fine;
}

HOST uint * GridImp::getCF_offset() const
{
    return this->gridInterface->cf.offset;
}

uint* GridImp::getFC_coarse() const
{
    return this->gridInterface->fc.coarse;
}

uint* GridImp::getFC_fine() const
{
    return this->gridInterface->fc.fine;
}

HOST uint * GridImp::getFC_offset() const
{
    return this->gridInterface->fc.offset;
}

void GridImp::getGridInterfaceIndices(uint* iCellCfc, uint* iCellCff, uint* iCellFcc, uint* iCellFcf) const
{
    getGridInterface(iCellCfc, this->gridInterface->cf.coarse, this->gridInterface->cf.numberOfEntries);
    getGridInterface(iCellCff, this->gridInterface->cf.fine, this->gridInterface->cf.numberOfEntries);
    getGridInterface(iCellFcc, this->gridInterface->fc.coarse, this->gridInterface->fc.numberOfEntries);
    getGridInterface(iCellFcf, this->gridInterface->fc.fine, this->gridInterface->fc.numberOfEntries);
}

void GridImp::getGridInterface(uint* gridInterfaceList, const uint* oldGridInterfaceList, uint size)
{
    for (uint i = 0; i < size; i++)
        gridInterfaceList[i] = oldGridInterfaceList[i] + 1; // + 1 for numbering shift between GridGenerator and VF_GPU
}

#define GEOFLUID 19
#define GEOSOLID 16

HOST void GridImp::getNodeValues(real *xCoords, real *yCoords, real *zCoords, uint *neighborX, uint *neighborY, uint *neighborZ, uint *neighborNegative, uint *geo) const
{
    xCoords[0] = 0;
    yCoords[0] = 0;
    zCoords[0] = 0;
    neighborX[0] = 0;
    neighborY[0] = 0;
    neighborZ[0] = 0;
    geo[0] = GEOSOLID;

    int nodeNumber = 0;
    for (uint i = 0; i < this->getSize(); i++)
    {
        if (this->sparseIndices[i] == -1)
            continue;

        real x, y, z;
        this->transIndexToCoords(i, x, y, z);

        // + 1 for numbering shift between GridGenerator and VF_GPU
        const uint neighborXIndex        = uint(this->neighborIndexX[i] + 1);
        const uint neighborYIndex        = uint(this->neighborIndexY[i] + 1);
        const uint neighborZIndex        = uint(this->neighborIndexZ[i] + 1);
        const uint neighborNegativeIndex = uint(this->neighborIndexNegative[i] + 1);

        const char type2 = this->field.getFieldEntry(i);

        const uint type = uint(this->field.isFluid(i) ? GEOFLUID : GEOSOLID);

        xCoords[nodeNumber + 1] = x;
        yCoords[nodeNumber + 1] = y;
        zCoords[nodeNumber + 1] = z;

        neighborX       [nodeNumber + 1] = neighborXIndex;
        neighborY       [nodeNumber + 1] = neighborYIndex;
        neighborZ       [nodeNumber + 1] = neighborZIndex;
        neighborNegative[nodeNumber + 1] = neighborNegativeIndex;

        geo[nodeNumber + 1] = type;
        nodeNumber++;
    }
}

void GridImp::print() const
{
    printf("min: (%2.4f, %2.4f, %2.4f), max: (%2.4f, %2.4f, %2.4f), size: %d, delta: %2.4f\n", startX, startY, startZ,
           endX, endY, endZ, size, delta);
    if(this->gridInterface)
        this->gridInterface->print();
}
