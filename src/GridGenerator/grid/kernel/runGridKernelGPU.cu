#include "runGridKernelGPU.cuh"

#include <GridGenerator/utilities/cuda/cudaDefines.h>
#include <GridGenerator/utilities/cuda/cudaKernelCall.h>
#include <GridGenerator/utilities/Launchparameter/LaunchParameter.cuh>

#include <GridGenerator/grid/Grid.cuh>
#include <GridGenerator/geometries/Geometry/Geometry.cuh>

GLOBAL void initalField(Grid grid);
GLOBAL void runMeshing(Grid grid, const Geometry geom);
GLOBAL void runKernelTomarkNodesToDeleteOutsideOfGeometry(Grid grid);
GLOBAL void markNodesToDeleteOutsideOfGeometry(Grid grid);
GLOBAL void findNeighborIndicesKernel(Grid grid);
GLOBAL void setOverlapNodesToInvalid(Grid grid, Grid finderGrid);

//////////////////////////////////////////////////////////////////////////

float runKernelInitalUniformGrid3d(const LaunchParameter& para, Grid &grid)
{
	return runKernel(initalField, para, grid);
}

GLOBAL void initalField(Grid grid)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
    {
        grid.setNeighborIndices(index);
        grid.matrixIndex[index] = index;
        grid.setFieldEntryToFluid(index);
    }
}

//////////////////////////////////////////////////////////////////////////

float runKernelToMesh(const LaunchParameter& para, Grid &grid, const Geometry &geom)
{
	return runKernel(runMeshing, para, grid, geom);
}

GLOBAL void runMeshing(Grid grid, const Geometry geom)
{
	unsigned int i = LaunchParameter::getGlobalIdx_1D_1D();
	if (i < geom.size)
		grid.meshTriangle(geom.triangles[i]);
}

//////////////////////////////////////////////////////////////////////////

float runKernelToMarkNodesToDeleteOutsideOfGeometry(const LaunchParameter& para, Grid &grid)
{
	return runKernel(markNodesToDeleteOutsideOfGeometry, para, grid);
}

GLOBAL void markNodesToDeleteOutsideOfGeometry(Grid grid)
{
    int numberOfEdgeNodes = grid.ny * grid.nz;
    unsigned int i = LaunchParameter::getGlobalIdx_1D_1D();

    if (i < numberOfEdgeNodes)
    {
        int x = 0;
        int y = i % grid.ny;
        int z = i / grid.ny;

        grid.setFieldEntryToSolid(grid.transCoordToIndex(x, y, z));

        while (grid.isFluid(i) && x < grid.nx - 1)
        {
            x += 1;
            grid.setFieldEntryToSolid(grid.transCoordToIndex(x, y, z));
        }
    }

}


//////////////////////////////////////////////////////////////////////////

float runKernelSetOverlapNodesToInvalid(const LaunchParameter& para, Grid &grid, Grid &finerGrid)
{
    return runKernel(setOverlapNodesToInvalid, para, grid, finerGrid);
}

GLOBAL void setOverlapNodesToInvalid(Grid grid, Grid finerGrid)
{
    const uint index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.createGridInterface(index, finerGrid);
}



/*#################################################################################*/
/*---------------------------------find invalid nodes------------------------------*/
/*---------------------------------------------------------------------------------*/
GLOBAL void setInvalidNodes(Grid grid, bool *foundInvalidNode);
/*---------------------------------------------------------------------------------*/

float runKernelSetToInvalid(const LaunchParameter& para, Grid &grid)
{
    bool* foundInvalidNode_d;
    bool* foundInvalidNode_h = new bool();
    *foundInvalidNode_h = true;
    CudaSafeCall( hipMalloc(&foundInvalidNode_d, sizeof(bool)));
    CudaCheckError();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    while (*foundInvalidNode_h)
    {
        *foundInvalidNode_h = false;
        CudaSafeCall(hipMemcpy(foundInvalidNode_d, foundInvalidNode_h, sizeof(bool), hipMemcpyHostToDevice));
        setInvalidNodes << <para.blocks, para.threads >> >(grid, foundInvalidNode_d);
        hipDeviceSynchronize();
        CudaCheckError();

        CudaSafeCall(hipMemcpy(foundInvalidNode_h, foundInvalidNode_d, sizeof(bool), hipMemcpyDeviceToHost));
    }

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    CudaCheckError();
    hipFree(foundInvalidNode_d);
    delete foundInvalidNode_h;

    return elapsedTime;
}


GLOBAL void setInvalidNodes(Grid grid, bool *foundInvalidNode)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.setInvalidNode(index, *foundInvalidNode);
}


/*#################################################################################*/

float runKernelFindIndices(const LaunchParameter& para, Grid &grid)
{
    return runKernel(findNeighborIndicesKernel, para, grid);
}

GLOBAL void findNeighborIndicesKernel(Grid grid)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.findNeighborIndex(index);
}
