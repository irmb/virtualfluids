#include "runGridKernelGPU.cuh"

#include <GridGenerator/utilities/cuda/cudaDefines.h>
#include <GridGenerator/utilities/cuda/cudaKernelCall.h>
#include <GridGenerator/utilities/Launchparameter/LaunchParameter.cuh>

#include <GridGenerator/grid/Grid.cuh>
#include <GridGenerator/geometries/Geometry/Geometry.cuh>

GLOBAL void initalField(Grid grid);
GLOBAL void runMeshing(Grid grid, const Geometry geom);

GLOBAL void findGridInterface(Grid grid, Grid finerGrid);
GLOBAL void runKernelTomarkNodesToDeleteOutsideOfGeometry(Grid grid);
GLOBAL void markNodesToDeleteOutsideOfGeometry(Grid grid);
GLOBAL void findNeighborIndicesKernel(Grid grid);
GLOBAL void setOverlapNodesToInvalid(Grid grid, Grid finderGrid);

//////////////////////////////////////////////////////////////////////////

float runKernelInitalUniformGrid3d(const LaunchParameter& para, Grid &grid)
{
	return runKernel(initalField, para, grid);
}

GLOBAL void initalField(Grid grid)
{
    uint index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.initalNodes(index);
}

//////////////////////////////////////////////////////////////////////////

float runKernelToMesh(const LaunchParameter& para, Grid &grid, const Geometry &geom)
{
	return runKernel(runMeshing, para, grid, geom);
}

GLOBAL void runMeshing(Grid grid, const Geometry geom)
{
	unsigned int i = LaunchParameter::getGlobalIdx_1D_1D();
	if (i < geom.size)
		grid.meshTriangle(geom.triangles[i]);
}

//////////////////////////////////////////////////////////////////////////

float runKernelToMarkNodesToDeleteOutsideOfGeometry(const LaunchParameter& para, Grid &grid)
{
	return runKernel(markNodesToDeleteOutsideOfGeometry, para, grid);
}

GLOBAL void markNodesToDeleteOutsideOfGeometry(Grid grid)
{
    int numberOfEdgeNodes = grid.ny * grid.nz;
    unsigned int i = LaunchParameter::getGlobalIdx_1D_1D();

    if (i < numberOfEdgeNodes)
    {
        int x = 0;
        int y = i % grid.ny;
        int z = i / grid.ny;

        grid.setFieldEntryToSolid(grid.transCoordToIndex(x, y, z));

        while (grid.isFluid(i) && x < grid.nx - 1)
        {
            x += 1;
            grid.setFieldEntryToSolid(grid.transCoordToIndex(x, y, z));
        }
    }

}


//////////////////////////////////////////////////////////////////////////

float runKernelSetOverlapNodesToInvalid(const LaunchParameter& para, Grid &grid, Grid &finerGrid)
{
    return runKernel(setOverlapNodesToInvalid, para, grid, finerGrid);
}

GLOBAL void setOverlapNodesToInvalid(Grid grid, Grid finerGrid)
{
    const uint index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.createGridInterface(index, finerGrid);
}



/*#################################################################################*/
/*---------------------------------find invalid nodes------------------------------*/
/*---------------------------------------------------------------------------------*/
GLOBAL void setInvalidNodes(Grid grid, bool *foundInvalidNode);
/*---------------------------------------------------------------------------------*/

float runKernelSetToInvalid(const LaunchParameter& para, Grid &grid)
{
    bool* foundInvalidNode_d;
    bool* foundInvalidNode_h = new bool();
    *foundInvalidNode_h = true;
    CudaSafeCall( hipMalloc(&foundInvalidNode_d, sizeof(bool)));
    CudaCheckError();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    while (*foundInvalidNode_h)
    {
        *foundInvalidNode_h = false;
        CudaSafeCall(hipMemcpy(foundInvalidNode_d, foundInvalidNode_h, sizeof(bool), hipMemcpyHostToDevice));
        setInvalidNodes << <para.blocks, para.threads >> >(grid, foundInvalidNode_d);
        hipDeviceSynchronize();
        CudaCheckError();

        CudaSafeCall(hipMemcpy(foundInvalidNode_h, foundInvalidNode_d, sizeof(bool), hipMemcpyDeviceToHost));
    }

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    CudaCheckError();
    hipFree(foundInvalidNode_d);
    delete foundInvalidNode_h;

    return elapsedTime;
}


GLOBAL void setInvalidNodes(Grid grid, bool *foundInvalidNode)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.setInvalidNode(index, *foundInvalidNode);
}


/*#################################################################################*/

float runKernelFindIndices(const LaunchParameter& para, Grid &grid)
{
    return runKernel(findNeighborIndicesKernel, para, grid);
}

GLOBAL void findNeighborIndicesKernel(Grid grid)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.findNeighborIndex(index);
}


/*#################################################################################*/

float runKernelToFindGridInterface(const LaunchParameter& para, Grid &grid, Grid &finerGrid)
{
    return runKernel(findGridInterface, para, grid, finerGrid);
}

GLOBAL void findGridInterface(Grid grid, Grid finerGrid)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.createGridInterface(index, finerGrid);
}
/*#################################################################################*/

GLOBAL void findNeighborsNewIndices(Grid grid);
float runKernelToFindNeighborsNewIndices(const LaunchParameter& para, Grid &grid)
{
    return runKernel(findNeighborsNewIndices, para, grid);
}

GLOBAL void findNeighborsNewIndices(Grid grid)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getSize())
        grid.findNeighborIndex(index);
}
/*#################################################################################*/

GLOBAL void findGridInterfaceNewIndicesFC(Grid grid);
GLOBAL void findGridInterfaceNewIndicesCF(Grid grid);
float runKernelToFindGridInterfaceNewIndices(const LaunchParameter& para, Grid &grid)
{
    runKernel(findGridInterfaceNewIndicesCF, para, grid);
    return runKernel(findGridInterfaceNewIndicesFC, para, grid);
}


GLOBAL void findGridInterfaceNewIndicesCF(Grid grid)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getNumberOfNodesCF())
        grid.findForGridInterfaceNewIndexCF(index);
}

GLOBAL void findGridInterfaceNewIndicesFC(Grid grid)
{
    unsigned int index = LaunchParameter::getGlobalIdx_2D_1D();
    if (index < grid.getNumberOfNodesFC())
        grid.findForGridInterfaceNewIndexFC(index);
}
