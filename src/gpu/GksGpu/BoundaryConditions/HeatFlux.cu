#include "hip/hip_runtime.h"
#include "HeatFlux.h"

#define _USE_MATH_DEFINES
#include <math.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "PointerDefinitions.h"
#include "Core/RealConstants.h"

#include "DataBase/DataBase.h"
#include "DataBase/DataBaseStruct.h"

#include "Definitions/MemoryAccessPattern.h"
#include "Definitions/PassiveScalar.h"

#include "FlowStateData/FlowStateData.cuh"
#include "FlowStateData/FlowStateDataConversion.cuh"
#include "FlowStateData/AccessDeviceData.cuh"

#include "FluxComputation/Moments.cuh"
#include "FluxComputation/ApplyFlux.cuh"
#include "FluxComputation/Transformation.cuh"
#include "FluxComputation/AssembleFlux.cuh"
#include "FluxComputation/ExpansionCoefficients.cuh"

#include "CudaUtility/CudaRunKernel.hpp"

namespace GksGpu{

//////////////////////////////////////////////////////////////////////////

__global__                 void boundaryConditionKernel  ( const DataBaseStruct dataBase, 
                                                           const HeatFluxStruct boundaryCondition, 
                                                           const Parameters parameters,
                                                           const uint startIndex,
                                                           const uint numberOfEntities );

__host__ __device__ inline void boundaryConditionFunction( const DataBaseStruct& dataBase, 
                                                           const HeatFluxStruct& boundaryCondition, 
                                                           const Parameters& parameters,
                                                           const uint startIndex,
                                                           const uint index );

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

void HeatFlux::runBoundaryConditionKernel(const SPtr<DataBase> dataBase, 
                                          const Parameters parameters, 
                                          const uint level)
{    
    CudaUtility::CudaGrid grid( this->numberOfCellsPerLevel[ level ], 32 );

    runKernel( boundaryConditionKernel,
               boundaryConditionFunction,
               dataBase->getDeviceType(), grid, 
               dataBase->toStruct(),
               this->toStruct(),
               parameters,
               this->startOfCellsPerLevel[ level ] );

    hipDeviceSynchronize();

    getLastCudaError("HeatFlux::runBoundaryConditionKernel( const SPtr<DataBase> dataBase, const Parameters parameters, const uint level )");
}

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

__global__ void boundaryConditionKernel(const DataBaseStruct dataBase, 
                                        const HeatFluxStruct boundaryCondition, 
                                        const Parameters parameters,
                                        const uint startIndex,
                                        const uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index >= numberOfEntities ) return;

    boundaryConditionFunction( dataBase, boundaryCondition, parameters, startIndex, index );
}

__host__ __device__ inline void boundaryConditionFunction(const DataBaseStruct& dataBase, 
                                                          const HeatFluxStruct& boundaryCondition, 
                                                          const Parameters& parameters,
                                                          const uint startIndex,
                                                          const uint index)
{
    uint ghostCellIdx  = boundaryCondition.ghostCells [ startIndex + index ];
    uint domainCellIdx = boundaryCondition.domainCells[ startIndex + index ];

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    PrimitiveVariables domainCellPrim;
    {
        ConservedVariables domainCellData;
        readCellData(domainCellIdx, dataBase, domainCellData);
        domainCellPrim = toPrimitiveVariables(domainCellData, parameters.K);
    }

    ConservedVariables flux;

    flux.rhoE = boundaryCondition.HRRPUA * parameters.dt * parameters.dx * parameters.dx;

    applyFluxToPosCell(dataBase, domainCellIdx, flux, 'z', parameters);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
}

HeatFlux::HeatFlux(SPtr<DataBase> dataBase, real  HRRPUA)
    : BoundaryCondition( dataBase )
{
    this->HRRPUA = HRRPUA;
}

bool HeatFlux::isWall()
{
    return true;
}

bool HeatFlux::isFluxBC()
{
    return false;
}

bool HeatFlux::secondCellsNeeded()
{
    return false;
}

} // namespace GksGpu

