#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __         
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |        
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |        
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |        
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____    
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|   
//      \    \  |    |   ________________________________________________________________    
//       \    \ |    |  |  ______________________________________________________________|   
//        \    \|    |  |  |         __          __     __     __     ______      _______    
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)   
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______    
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/   
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can 
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of 
//  the License, or (at your option) any later version.
//  
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT 
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or 
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License 
//  for more details.
//  
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file IsothermalWall.cu
//! \ingroup BoundaryCondition
//! \author Stephan Lenz
//=======================================================================================
#include "IsothermalWall.h"

#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "PointerDefinitions.h"
#include "Core/RealConstants.h"

#include "DataBase/DataBase.h"
#include "DataBase/DataBaseStruct.h"

#include "Definitions/MemoryAccessPattern.h"
#include "Definitions/PassiveScalar.h"

#include "FlowStateData/FlowStateData.cuh"
#include "FlowStateData/FlowStateDataConversion.cuh"
#include "FlowStateData/AccessDeviceData.cuh"

#include "CudaUtility/CudaRunKernel.hpp"

//////////////////////////////////////////////////////////////////////////

__global__                 void boundaryConditionKernel  ( const DataBaseStruct dataBase, 
                                                           const IsothermalWallStruct boundaryCondition, 
                                                           const Parameters parameters,
                                                           const uint startIndex,
                                                           const uint numberOfEntities );

__host__ __device__ inline void boundaryConditionFunction( const DataBaseStruct& dataBase, 
                                                           const IsothermalWallStruct& boundaryCondition, 
                                                           const Parameters& parameters,
                                                           const uint startIndex,
                                                           const uint index );

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

void IsothermalWall::runBoundaryConditionKernel(const SPtr<DataBase> dataBase, 
                                                const Parameters parameters, 
                                                const uint level)
{    
    CudaUtility::CudaGrid grid( this->numberOfCellsPerLevel[ level ], 32 );

    runKernel( boundaryConditionKernel,
               boundaryConditionFunction,
               dataBase->getDeviceType(), grid, 
               dataBase->toStruct(),
               this->toStruct(),
               parameters,
               this->startOfCellsPerLevel[ level ] );

    hipDeviceSynchronize();

    getLastCudaError("IsothermalWall::runBoundaryConditionKernel( const SPtr<DataBase> dataBase, const Parameters parameters, const uint level )");
}

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

__global__ void boundaryConditionKernel(const DataBaseStruct dataBase, 
                                        const IsothermalWallStruct boundaryCondition, 
                                        const Parameters parameters,
                                        const uint startIndex,
                                        const uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index >= numberOfEntities ) return;

    boundaryConditionFunction( dataBase, boundaryCondition, parameters, startIndex, index );
}

__host__ __device__ inline void boundaryConditionFunction(const DataBaseStruct& dataBase, 
                                                          const IsothermalWallStruct& boundaryCondition, 
                                                          const Parameters& parameters,
                                                          const uint startIndex,
                                                          const uint index)
{
    uint ghostCellIdx  = boundaryCondition.ghostCells [ startIndex + index ];
    uint domainCellIdx = boundaryCondition.domainCells[ startIndex + index ];
    uint secondCellIdx = boundaryCondition.secondCells[ startIndex + index ];

    PrimitiveVariables ghostCellPrim;
    {
        PrimitiveVariables domainCellPrim;
        PrimitiveVariables secondCellPrim;

        {
            ConservedVariables domainCellData;
            readCellData( domainCellIdx, dataBase, domainCellData );
            domainCellPrim = toPrimitiveVariables( domainCellData, parameters.K );

            ConservedVariables secondCellData;
            if( secondCellIdx != INVALID_INDEX ){
                readCellData( secondCellIdx, dataBase, secondCellData );
                secondCellPrim = toPrimitiveVariables( secondCellData, parameters.K );
            }
        }

        ghostCellPrim.U      = c2o1 * boundaryCondition.velocity.x - domainCellPrim.U;
        ghostCellPrim.V      = c2o1 * boundaryCondition.velocity.y - domainCellPrim.V;
        ghostCellPrim.W      = c2o1 * boundaryCondition.velocity.z - domainCellPrim.W;
        ghostCellPrim.lambda = c2o1 * boundaryCondition.lambda     - domainCellPrim.lambda;
    #ifdef USE_PASSIVE_SCALAR
        ghostCellPrim.S_1    = /*two * boundaryCondition.S_1 -*/ domainCellPrim.S_1;
        ghostCellPrim.S_2    = /*two * boundaryCondition.S_2 -*/ domainCellPrim.S_2;
    #endif // USE_PASSIVE_SCALAR


        if( boundaryCondition.useSecondCells && secondCellIdx != INVALID_INDEX ){
            // linear extrapolation
            real p1 = c1o2 * domainCellPrim.rho / domainCellPrim.lambda;
            real p2 = c1o2 * secondCellPrim.rho / secondCellPrim.lambda;

            ghostCellPrim.rho = c2o1 * ( c2o1 * p1 - p2 ) * ghostCellPrim.lambda;
        }
        else{
            // constant extrapolation
            real p = c1o2 * domainCellPrim.rho / domainCellPrim.lambda;

            ghostCellPrim.rho = c2o1 * p * ghostCellPrim.lambda;
        }
    }

    {
        ConservedVariables ghostCons = toConservedVariables( ghostCellPrim, parameters.K );

        writeCellData( ghostCellIdx, dataBase, ghostCons );
    }
}

IsothermalWall::IsothermalWall(SPtr<DataBase> dataBase, Vec3 velocity, real lambda, bool useSecondCells, real S_1, real S_2)
    : BoundaryCondition( dataBase )
{
    this->velocity       = velocity;
    this->lambda         = lambda;
    this->S_1            = S_1;
    this->S_2            = S_2;
    this->useSecondCells = useSecondCells;
}

bool IsothermalWall::isWall()
{
    return true;
}

bool IsothermalWall::secondCellsNeeded()
{
    return true;
}

