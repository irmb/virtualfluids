#include "hip/hip_runtime.h"
#include "TurbulenceAnalyzer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cmath>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#include <iomanip>

#include "Core/Logger/Logger.h"

#include "DataBase/DataBase.h"

#include "Parameters/Parameters.h"

#include "FlowStateData/FlowStateData.cuh"
#include "FlowStateData/FlowStateDataConversion.cuh"
#include "FlowStateData/AccessDeviceData.cuh"

#include "CudaUtility/CudaRunKernel.hpp"

namespace GksGpu {

__global__                 void turbulenceKernel  ( DataBaseStruct dataBase, TurbulenceAnalyzerStruct turbulenceAnalyzer, Parameters parameters, uint startIndex, uint numberOfEntities );

__host__ __device__ inline void turbulenceFunction( DataBaseStruct dataBase, TurbulenceAnalyzerStruct turbulenceAnalyzer, Parameters parameters, uint startIndex, uint index );

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

bool TurbulenceAnalyzer::run(uint iter, Parameters parameters)
{
    if( iter < this->analyzeStartIter ) return false;

    CudaUtility::CudaGrid grid( dataBase->numberOfCells, 32 );

    runKernel( turbulenceKernel,
               turbulenceFunction,
               dataBase->getDeviceType(), grid, 
               dataBase->toStruct(),
               this->toStruct(),
               parameters,
               0 );

    getLastCudaError("TurbulenceAnalyzer::run(uint iter, Parameters parameters)");

    this->counter++;

    return true;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void turbulenceKernel(DataBaseStruct dataBase, TurbulenceAnalyzerStruct turbulenceAnalyzer, Parameters parameters, uint startIndex, uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index >= numberOfEntities ) return;

    turbulenceFunction( dataBase, turbulenceAnalyzer, parameters, startIndex, index );
}

__host__ __device__ void turbulenceFunction(DataBaseStruct dataBase, TurbulenceAnalyzerStruct turbulenceAnalyzer, Parameters parameters, uint startIndex, uint index)
{
    uint cellIndex = startIndex + index;

    //////////////////////////////////////////////////////////////////////////

    ConservedVariables cons;

    readCellData(cellIndex, dataBase, cons);

    PrimitiveVariables prim = toPrimitiveVariables(cons, parameters.K);

    //////////////////////////////////////////////////////////////////////////

    if( turbulenceAnalyzer.U  ) turbulenceAnalyzer.U [ cellIndex ] += prim.U;
    if( turbulenceAnalyzer.V  ) turbulenceAnalyzer.V [ cellIndex ] += prim.V;
    if( turbulenceAnalyzer.W  ) turbulenceAnalyzer.W [ cellIndex ] += prim.W;

    if( turbulenceAnalyzer.UU ) turbulenceAnalyzer.UU[ cellIndex ] += prim.U * prim.U;
    if( turbulenceAnalyzer.VV ) turbulenceAnalyzer.VV[ cellIndex ] += prim.V * prim.V;
    if( turbulenceAnalyzer.WW ) turbulenceAnalyzer.WW[ cellIndex ] += prim.W * prim.W;

    if( turbulenceAnalyzer.UV ) turbulenceAnalyzer.UV[ cellIndex ] += prim.U * prim.V;
    if( turbulenceAnalyzer.UW ) turbulenceAnalyzer.UW[ cellIndex ] += prim.U * prim.W;
    if( turbulenceAnalyzer.VW ) turbulenceAnalyzer.VW[ cellIndex ] += prim.V * prim.W;

#ifdef USE_PASSIVE_SCALAR
    if( turbulenceAnalyzer.T  ) turbulenceAnalyzer.T [ cellIndex ] += getT(prim);
#else
    if( turbulenceAnalyzer.T  ) turbulenceAnalyzer.T [ cellIndex ] +=   c1o1 / prim.lambda;
#endif

    if( turbulenceAnalyzer.TT ) turbulenceAnalyzer.TT[ cellIndex ] += ( c1o1 / prim.lambda ) * ( c1o1 / prim.lambda );
    if( turbulenceAnalyzer.p  ) turbulenceAnalyzer.p [ cellIndex ] += c1o2 * prim.rho / prim.lambda;

    //////////////////////////////////////////////////////////////////////////
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

TurbulenceAnalyzer::~TurbulenceAnalyzer()
{
    this->free();
}

TurbulenceAnalyzer::TurbulenceAnalyzer(SPtr<DataBase> dataBase, uint analyzeStartIter)
    : U ( nullptr ),
      V ( nullptr ),
      W ( nullptr ),
      UU( nullptr ),
      VV( nullptr ),
      WW( nullptr ),
      UV( nullptr ),
      UW( nullptr ),
      VW( nullptr ),
      T ( nullptr ),
      TT( nullptr ),
      p ( nullptr ),
      collect_U ( true  ),
      collect_V ( true  ),
      collect_W ( true  ),
      collect_UU( false ),
      collect_VV( false ),
      collect_WW( false ),
      collect_UV( false ),
      collect_UW( false ),
      collect_VW( false ),
      collect_T ( true  ),
      collect_TT( false ),
      collect_p ( false )
{
    this->dataBase = dataBase;

    this->analyzeStartIter = analyzeStartIter;

    this->counter = 0;

    this->allocate();
}

void TurbulenceAnalyzer::free()
{
    if( this->U  ) checkCudaErrors( hipFree ( this->U  ) );
    if( this->V  ) checkCudaErrors( hipFree ( this->V  ) );
    if( this->W  ) checkCudaErrors( hipFree ( this->W  ) );
    if( this->UU ) checkCudaErrors( hipFree ( this->UU ) );
    if( this->VV ) checkCudaErrors( hipFree ( this->VV ) );
    if( this->WW ) checkCudaErrors( hipFree ( this->WW ) );
    if( this->UV ) checkCudaErrors( hipFree ( this->UV ) );
    if( this->UW ) checkCudaErrors( hipFree ( this->UW ) );
    if( this->VW ) checkCudaErrors( hipFree ( this->VW ) );
    if( this->T  ) checkCudaErrors( hipFree ( this->T  ) );
    if( this->TT ) checkCudaErrors( hipFree ( this->TT ) );
    if( this->p  ) checkCudaErrors( hipFree ( this->p  ) );

    h_U.clear ( );
    h_V.clear ( );
    h_W.clear ( );
    h_UU.clear( );
    h_VV.clear( );
    h_WW.clear( );
    h_UV.clear( );
    h_UW.clear( );
    h_VW.clear( );
    h_T.clear ( );
    h_TT.clear( );
    h_p.clear ( );
}

void TurbulenceAnalyzer::allocate()
{
    this->free();

    if( collect_U  ) checkCudaErrors( hipMalloc ( &this->U , sizeof(real) * dataBase->numberOfCells ) );
    if( collect_V  ) checkCudaErrors( hipMalloc ( &this->V , sizeof(real) * dataBase->numberOfCells ) );
    if( collect_W  ) checkCudaErrors( hipMalloc ( &this->W , sizeof(real) * dataBase->numberOfCells ) );
    if( collect_UU ) checkCudaErrors( hipMalloc ( &this->UU, sizeof(real) * dataBase->numberOfCells ) );
    if( collect_VV ) checkCudaErrors( hipMalloc ( &this->VV, sizeof(real) * dataBase->numberOfCells ) );
    if( collect_WW ) checkCudaErrors( hipMalloc ( &this->WW, sizeof(real) * dataBase->numberOfCells ) );
    if( collect_UV ) checkCudaErrors( hipMalloc ( &this->UV, sizeof(real) * dataBase->numberOfCells ) );
    if( collect_UW ) checkCudaErrors( hipMalloc ( &this->UW, sizeof(real) * dataBase->numberOfCells ) );
    if( collect_VW ) checkCudaErrors( hipMalloc ( &this->VW, sizeof(real) * dataBase->numberOfCells ) );
    if( collect_T  ) checkCudaErrors( hipMalloc ( &this->T , sizeof(real) * dataBase->numberOfCells ) );
    if( collect_TT ) checkCudaErrors( hipMalloc ( &this->TT, sizeof(real) * dataBase->numberOfCells ) );
    if( collect_p  ) checkCudaErrors( hipMalloc ( &this->p , sizeof(real) * dataBase->numberOfCells ) );

    if( collect_U  ) h_U.resize ( dataBase->numberOfCells );
    if( collect_V  ) h_V.resize ( dataBase->numberOfCells ); 
    if( collect_W  ) h_W.resize ( dataBase->numberOfCells );
    if( collect_UU ) h_UU.resize( dataBase->numberOfCells );
    if( collect_VV ) h_VV.resize( dataBase->numberOfCells );
    if( collect_WW ) h_WW.resize( dataBase->numberOfCells );
    if( collect_UV ) h_UV.resize( dataBase->numberOfCells );
    if( collect_UW ) h_UW.resize( dataBase->numberOfCells );
    if( collect_VW ) h_VW.resize( dataBase->numberOfCells );
    if( collect_T  ) h_T.resize ( dataBase->numberOfCells );
    if( collect_TT ) h_TT.resize( dataBase->numberOfCells );
    if( collect_p  ) h_p.resize ( dataBase->numberOfCells );
}

void TurbulenceAnalyzer::writeRestartFile(std::string filename)
{
    this->download(false);

    //////////////////////////////////////////////////////////////////////////

    filename += ".rst";

    *logging::out << logging::Logger::INFO_HIGH << "Writing restart file " << filename << " ... ";

    std::ofstream file;

	file.open( filename.c_str(), std::ios::binary );

	if (!file.is_open()) {
		throw std::runtime_error("\nFile cannot be opened.\n\nERROR!\n\n\n");
        return;
	}

    //////////////////////////////////////////////////////////////////////////

    file.write( (char*) &this->counter, sizeof( uint ) );

    file.write( (char*) &dataBase->numberOfLevels, sizeof( uint ) );
    file.write( (char*) &dataBase->numberOfCells,  sizeof( uint ) );
    file.write( (char*) &dataBase->numberOfFaces,  sizeof( uint ) );

    file.write( (char*) &this->collect_U , sizeof( bool ) );
    file.write( (char*) &this->collect_V , sizeof( bool ) );
    file.write( (char*) &this->collect_W , sizeof( bool ) );
    file.write( (char*) &this->collect_UU, sizeof( bool ) );
    file.write( (char*) &this->collect_VV, sizeof( bool ) );
    file.write( (char*) &this->collect_WW, sizeof( bool ) );
    file.write( (char*) &this->collect_UV, sizeof( bool ) );
    file.write( (char*) &this->collect_UW, sizeof( bool ) );
    file.write( (char*) &this->collect_VW, sizeof( bool ) );
    file.write( (char*) &this->collect_T , sizeof( bool ) );
    file.write( (char*) &this->collect_TT, sizeof( bool ) );
    file.write( (char*) &this->collect_p , sizeof( bool ) );

    if( collect_U  ) file.write( (char*) this->h_U.data() , dataBase->numberOfCells * sizeof( real ) );
    if( collect_V  ) file.write( (char*) this->h_V.data() , dataBase->numberOfCells * sizeof( real ) );
    if( collect_W  ) file.write( (char*) this->h_W.data() , dataBase->numberOfCells * sizeof( real ) );
    if( collect_UU ) file.write( (char*) this->h_UU.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_VV ) file.write( (char*) this->h_VV.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_WW ) file.write( (char*) this->h_WW.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_UV ) file.write( (char*) this->h_UV.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_UW ) file.write( (char*) this->h_UW.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_VW ) file.write( (char*) this->h_VW.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_T  ) file.write( (char*) this->h_T.data() , dataBase->numberOfCells * sizeof( real ) );
    if( collect_TT ) file.write( (char*) this->h_TT.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_p  ) file.write( (char*) this->h_p.data() , dataBase->numberOfCells * sizeof( real ) );

    //////////////////////////////////////////////////////////////////////////

    file.close();

    *logging::out << logging::Logger::INFO_HIGH << "done!\n";
}

void TurbulenceAnalyzer::readRestartFile(std::string filename)
{
    filename += ".rst";

    *logging::out << logging::Logger::INFO_HIGH << "Reading restart file " << filename << " ... ";
	
    std::ifstream file;

	file.open( filename.c_str(), std::ios::binary );

	if (!file.is_open()) {
		throw std::runtime_error("\nFile cannot be opened.\n\nERROR!\n\n\n");
        return;
	}

    //////////////////////////////////////////////////////////////////////////

    file.read( (char*) &this->counter, sizeof( uint ) );

    uint numberOfLevelsRead;
    uint numberOfCellsRead;
    uint numberOfFacesRead;
    
    file.read( (char*) &numberOfLevelsRead, sizeof( uint ) );
    file.read( (char*) &numberOfCellsRead,  sizeof( uint ) );
    file.read( (char*) &numberOfFacesRead,  sizeof( uint ) );

    if( numberOfLevelsRead != dataBase->numberOfLevels ||
        numberOfCellsRead  != dataBase->numberOfCells  ||
        numberOfFacesRead  != dataBase->numberOfFaces  ){
    
        *logging::out << logging::Logger::INFO_HIGH << "\n";
        *logging::out << logging::Logger::INFO_HIGH << "Levels: " << numberOfLevelsRead << " vs. " << dataBase->numberOfLevels << "\n";
        *logging::out << logging::Logger::INFO_HIGH << "Cells:  " << numberOfCellsRead  << " vs. " << dataBase->numberOfCells  << "\n";
        *logging::out << logging::Logger::INFO_HIGH << "Faces:  " << numberOfFacesRead  << " vs. " << dataBase->numberOfFaces  << "\n";

        file.close();

        throw std::runtime_error("\nERROR: Restart file does not match current setup");
    }

    //////////////////////////////////////////////////////////////////////////

    file.read( (char*) &this->collect_U , sizeof( bool ) );
    file.read( (char*) &this->collect_V , sizeof( bool ) );
    file.read( (char*) &this->collect_W , sizeof( bool ) );
    file.read( (char*) &this->collect_UU, sizeof( bool ) );
    file.read( (char*) &this->collect_VV, sizeof( bool ) );
    file.read( (char*) &this->collect_WW, sizeof( bool ) );
    file.read( (char*) &this->collect_UV, sizeof( bool ) );
    file.read( (char*) &this->collect_UW, sizeof( bool ) );
    file.read( (char*) &this->collect_VW, sizeof( bool ) );
    file.read( (char*) &this->collect_T , sizeof( bool ) );
    file.read( (char*) &this->collect_TT, sizeof( bool ) );
    file.read( (char*) &this->collect_p , sizeof( bool ) );

    if( collect_U  ) file.read( (char*) this->h_U.data() , dataBase->numberOfCells * sizeof( real ) );
    if( collect_V  ) file.read( (char*) this->h_V.data() , dataBase->numberOfCells * sizeof( real ) );
    if( collect_W  ) file.read( (char*) this->h_W.data() , dataBase->numberOfCells * sizeof( real ) );
    if( collect_UU ) file.read( (char*) this->h_UU.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_VV ) file.read( (char*) this->h_VV.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_WW ) file.read( (char*) this->h_WW.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_UV ) file.read( (char*) this->h_UV.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_UW ) file.read( (char*) this->h_UW.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_VW ) file.read( (char*) this->h_VW.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_T  ) file.read( (char*) this->h_T.data() , dataBase->numberOfCells * sizeof( real ) );
    if( collect_TT ) file.read( (char*) this->h_TT.data(), dataBase->numberOfCells * sizeof( real ) );
    if( collect_p  ) file.read( (char*) this->h_p.data() , dataBase->numberOfCells * sizeof( real ) );

    //////////////////////////////////////////////////////////////////////////

    file.close();

    this->upload();

    *logging::out << logging::Logger::INFO_HIGH << "done!\n";
}

TurbulenceAnalyzerStruct TurbulenceAnalyzer::toStruct()
{
    TurbulenceAnalyzerStruct turbulenceAnalyzer;

    turbulenceAnalyzer.U  = this->U;
    turbulenceAnalyzer.V  = this->V;
    turbulenceAnalyzer.W  = this->W;

    turbulenceAnalyzer.UU = this->UU;
    turbulenceAnalyzer.VV = this->VV;
    turbulenceAnalyzer.WW = this->WW;

    turbulenceAnalyzer.UV = this->UV;
    turbulenceAnalyzer.UW = this->UW;
    turbulenceAnalyzer.VW = this->VW;

    turbulenceAnalyzer.T  = this->T;
    turbulenceAnalyzer.TT = this->TT;
    turbulenceAnalyzer.p  = this->p;

    return turbulenceAnalyzer;
}

void TurbulenceAnalyzer::download(bool normalize)
{
    if( collect_U  ) checkCudaErrors( hipMemcpy( this->h_U.data() , this->U , sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_V  ) checkCudaErrors( hipMemcpy( this->h_V.data() , this->V , sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_W  ) checkCudaErrors( hipMemcpy( this->h_W.data() , this->W , sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_UU ) checkCudaErrors( hipMemcpy( this->h_UU.data(), this->UU, sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_VV ) checkCudaErrors( hipMemcpy( this->h_VV.data(), this->VV, sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_WW ) checkCudaErrors( hipMemcpy( this->h_WW.data(), this->WW, sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_UV ) checkCudaErrors( hipMemcpy( this->h_UV.data(), this->UV, sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_UW ) checkCudaErrors( hipMemcpy( this->h_UW.data(), this->UW, sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_VW ) checkCudaErrors( hipMemcpy( this->h_VW.data(), this->VW, sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_T  ) checkCudaErrors( hipMemcpy( this->h_T.data() , this->T , sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_TT ) checkCudaErrors( hipMemcpy( this->h_TT.data(), this->TT, sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );
    if( collect_p  ) checkCudaErrors( hipMemcpy( this->h_p.data() , this->p , sizeof(real) * dataBase->numberOfCells, hipMemcpyDeviceToHost ) );

    if(normalize)
    {
        for( uint cellIndex = 0; cellIndex < dataBase->numberOfCells; cellIndex++ )
        {
            if( collect_U  ) this->h_U [ cellIndex ] /= real(this->counter);
            if( collect_V  ) this->h_V [ cellIndex ] /= real(this->counter);
            if( collect_W  ) this->h_W [ cellIndex ] /= real(this->counter);
            if( collect_UU ) this->h_UU[ cellIndex ] /= real(this->counter);
            if( collect_VV ) this->h_VV[ cellIndex ] /= real(this->counter);
            if( collect_WW ) this->h_WW[ cellIndex ] /= real(this->counter);
            if( collect_UV ) this->h_UV[ cellIndex ] /= real(this->counter);
            if( collect_UW ) this->h_UW[ cellIndex ] /= real(this->counter);
            if( collect_VW ) this->h_VW[ cellIndex ] /= real(this->counter);
            if( collect_T  ) this->h_T [ cellIndex ] /= real(this->counter);
            if( collect_TT ) this->h_TT[ cellIndex ] /= real(this->counter);
            if( collect_p  ) this->h_p [ cellIndex ] /= real(this->counter);

            if( collect_UU ) this->h_UU[ cellIndex ] -= this->h_U[ cellIndex ] * this->h_U[ cellIndex ];
            if( collect_VV ) this->h_VV[ cellIndex ] -= this->h_V[ cellIndex ] * this->h_V[ cellIndex ];
            if( collect_WW ) this->h_WW[ cellIndex ] -= this->h_W[ cellIndex ] * this->h_W[ cellIndex ];

            if( collect_UV ) this->h_UV[ cellIndex ] -= this->h_U[ cellIndex ] * this->h_V[ cellIndex ];
            if( collect_UW ) this->h_UW[ cellIndex ] -= this->h_U[ cellIndex ] * this->h_W[ cellIndex ];
            if( collect_VW ) this->h_VW[ cellIndex ] -= this->h_V[ cellIndex ] * this->h_W[ cellIndex ];
        
            if( collect_TT ) this->h_TT[ cellIndex ] -= this->h_T[ cellIndex ] * this->h_T[ cellIndex ];
        }
    }
}

void TurbulenceAnalyzer::upload()
{
    if( collect_U  ) checkCudaErrors( hipMemcpy( this->U , this->h_U.data() , sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_V  ) checkCudaErrors( hipMemcpy( this->V , this->h_V.data() , sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_W  ) checkCudaErrors( hipMemcpy( this->W , this->h_W.data() , sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_UU ) checkCudaErrors( hipMemcpy( this->UU, this->h_UU.data(), sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_VV ) checkCudaErrors( hipMemcpy( this->VV, this->h_VV.data(), sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_WW ) checkCudaErrors( hipMemcpy( this->WW, this->h_WW.data(), sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_UV ) checkCudaErrors( hipMemcpy( this->UV, this->h_UV.data(), sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_UW ) checkCudaErrors( hipMemcpy( this->UW, this->h_UW.data(), sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_VW ) checkCudaErrors( hipMemcpy( this->VW, this->h_VW.data(), sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_T  ) checkCudaErrors( hipMemcpy( this->T , this->h_T.data() , sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_TT ) checkCudaErrors( hipMemcpy( this->TT, this->h_TT.data(), sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
    if( collect_p  ) checkCudaErrors( hipMemcpy( this->p , this->h_p.data() , sizeof(real) * dataBase->numberOfCells, hipMemcpyHostToDevice ) );
}

} // namespace GksGpu


