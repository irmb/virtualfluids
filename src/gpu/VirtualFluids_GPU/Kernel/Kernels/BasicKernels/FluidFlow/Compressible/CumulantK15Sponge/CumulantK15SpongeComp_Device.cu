#include "hip/hip_runtime.h"
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include "Core/RealConstants.h"
#include "math.h"

extern "C" __global__ void LB_Kernel_CumulantK15SpongeComp(real omegaIn,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* coordX,
	real* coordY,
	real* coordZ,
	real* DDStart,
	int size_Mat,
	bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if ((BC != GEO_SOLID) && (BC != GEO_VOID))
		{
			Distributions27 D;
			if (EvenOrOdd == true)
			{
				D.f[dirE] = &DDStart[dirE   *size_Mat];
				D.f[dirW] = &DDStart[dirW   *size_Mat];
				D.f[dirN] = &DDStart[dirN   *size_Mat];
				D.f[dirS] = &DDStart[dirS   *size_Mat];
				D.f[dirT] = &DDStart[dirT   *size_Mat];
				D.f[dirB] = &DDStart[dirB   *size_Mat];
				D.f[dirNE] = &DDStart[dirNE  *size_Mat];
				D.f[dirSW] = &DDStart[dirSW  *size_Mat];
				D.f[dirSE] = &DDStart[dirSE  *size_Mat];
				D.f[dirNW] = &DDStart[dirNW  *size_Mat];
				D.f[dirTE] = &DDStart[dirTE  *size_Mat];
				D.f[dirBW] = &DDStart[dirBW  *size_Mat];
				D.f[dirBE] = &DDStart[dirBE  *size_Mat];
				D.f[dirTW] = &DDStart[dirTW  *size_Mat];
				D.f[dirTN] = &DDStart[dirTN  *size_Mat];
				D.f[dirBS] = &DDStart[dirBS  *size_Mat];
				D.f[dirBN] = &DDStart[dirBN  *size_Mat];
				D.f[dirTS] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirTNE] = &DDStart[dirTNE *size_Mat];
				D.f[dirTSW] = &DDStart[dirTSW *size_Mat];
				D.f[dirTSE] = &DDStart[dirTSE *size_Mat];
				D.f[dirTNW] = &DDStart[dirTNW *size_Mat];
				D.f[dirBNE] = &DDStart[dirBNE *size_Mat];
				D.f[dirBSW] = &DDStart[dirBSW *size_Mat];
				D.f[dirBSE] = &DDStart[dirBSE *size_Mat];
				D.f[dirBNW] = &DDStart[dirBNW *size_Mat];
			}
			else
			{
				D.f[dirW] = &DDStart[dirE   *size_Mat];
				D.f[dirE] = &DDStart[dirW   *size_Mat];
				D.f[dirS] = &DDStart[dirN   *size_Mat];
				D.f[dirN] = &DDStart[dirS   *size_Mat];
				D.f[dirB] = &DDStart[dirT   *size_Mat];
				D.f[dirT] = &DDStart[dirB   *size_Mat];
				D.f[dirSW] = &DDStart[dirNE  *size_Mat];
				D.f[dirNE] = &DDStart[dirSW  *size_Mat];
				D.f[dirNW] = &DDStart[dirSE  *size_Mat];
				D.f[dirSE] = &DDStart[dirNW  *size_Mat];
				D.f[dirBW] = &DDStart[dirTE  *size_Mat];
				D.f[dirTE] = &DDStart[dirBW  *size_Mat];
				D.f[dirTW] = &DDStart[dirBE  *size_Mat];
				D.f[dirBE] = &DDStart[dirTW  *size_Mat];
				D.f[dirBS] = &DDStart[dirTN  *size_Mat];
				D.f[dirTN] = &DDStart[dirBS  *size_Mat];
				D.f[dirTS] = &DDStart[dirBN  *size_Mat];
				D.f[dirBN] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirBSW] = &DDStart[dirTNE *size_Mat];
				D.f[dirBNE] = &DDStart[dirTSW *size_Mat];
				D.f[dirBNW] = &DDStart[dirTSE *size_Mat];
				D.f[dirBSE] = &DDStart[dirTNW *size_Mat];
				D.f[dirTSW] = &DDStart[dirBNE *size_Mat];
				D.f[dirTNE] = &DDStart[dirBSW *size_Mat];
				D.f[dirTNW] = &DDStart[dirBSE *size_Mat];
				D.f[dirTSE] = &DDStart[dirBNW *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			//unsigned int kzero= k;
			//unsigned int ke   = k;
			unsigned int kw = neighborX[k];
			//unsigned int kn   = k;
			unsigned int ks = neighborY[k];
			//unsigned int kt   = k;
			unsigned int kb = neighborZ[k];
			unsigned int ksw = neighborY[kw];
			//unsigned int kne  = k;
			//unsigned int kse  = ks;
			//unsigned int knw  = kw;
			unsigned int kbw = neighborZ[kw];
			//unsigned int kte  = k;
			//unsigned int kbe  = kb;
			//unsigned int ktw  = kw;
			unsigned int kbs = neighborZ[ks];
			//unsigned int ktn  = k;
			//unsigned int kbn  = kb;
			//unsigned int kts  = ks;
			//unsigned int ktse = ks;
			//unsigned int kbnw = kbw;
			//unsigned int ktnw = kw;
			//unsigned int kbse = kbs;
			//unsigned int ktsw = ksw;
			//unsigned int kbne = kb;
			//unsigned int ktne = k;
			unsigned int kbsw = neighborZ[ksw];
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[dirE])[k];
			real mfabb = (D.f[dirW])[kw];
			real mfbcb = (D.f[dirN])[k];
			real mfbab = (D.f[dirS])[ks];
			real mfbbc = (D.f[dirT])[k];
			real mfbba = (D.f[dirB])[kb];
			real mfccb = (D.f[dirNE])[k];
			real mfaab = (D.f[dirSW])[ksw];
			real mfcab = (D.f[dirSE])[ks];
			real mfacb = (D.f[dirNW])[kw];
			real mfcbc = (D.f[dirTE])[k];
			real mfaba = (D.f[dirBW])[kbw];
			real mfcba = (D.f[dirBE])[kb];
			real mfabc = (D.f[dirTW])[kw];
			real mfbcc = (D.f[dirTN])[k];
			real mfbaa = (D.f[dirBS])[kbs];
			real mfbca = (D.f[dirBN])[kb];
			real mfbac = (D.f[dirTS])[ks];
			real mfbbb = (D.f[dirZERO])[k];
			real mfccc = (D.f[dirTNE])[k];
			real mfaac = (D.f[dirTSW])[ksw];
			real mfcac = (D.f[dirTSE])[ks];
			real mfacc = (D.f[dirTNW])[kw];
			real mfcca = (D.f[dirBNE])[kb];
			real mfaaa = (D.f[dirBSW])[kbsw];
			real mfcaa = (D.f[dirBSE])[kbs];
			real mfaca = (D.f[dirBNW])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			real drho = ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;

			real rho = c1o1 + drho;
			////////////////////////////////////////////////////////////////////////////////////
			real vvx = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
				(((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
				(mfcbb - mfabb)) / rho;
			real vvy = ((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
				(((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
				(mfbcb - mfbab)) / rho;
			real vvz = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
				(((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
				(mfbbc - mfbba)) / rho;
			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			real fx = c0o1; //0.000000005;//(two/1600000.0) / 120.0; //zero;
			real fy = c0o1;
			real fz = c0o1;
			vvx += fx;
			vvy += fy;
			vvz += fz;
			////////////////////////////////////////////////////////////////////////////////////
			real oMdrho = c1o1; // comp special
							   ////////////////////////////////////////////////////////////////////////////////////
			real m0, m1, m2;
			real vx2 = vvx * vvx;
			real vy2 = vvy * vvy;
			real vz2 = vvz * vvz;
			////////////////////////////////////////////////////////////////////////////////////
			real wadjust;
			real qudricLimit = 0.01f;
			////////////////////////////////////////////////////////////////////////////////////
			//sponge layer inflow
			real startXsponge = 1507.0f;//120.995703125;
			real endXsponge = 1537.0f;//120.987890625;
			real sizeSponge = endXsponge - startXsponge;
			real factor = c1o1;
			real omega = factor * omegaIn;
			if (coordX[k] > startXsponge) {
				factor = (((endXsponge - coordX[k]) / sizeSponge) * c1o2) + c1o2;
				omega = factor * omegaIn;
			}
			////////////////////////////////////////////////////////////////////////////////////
			//sponge layer outflow
			endXsponge = 30.0f;
			if (coordX[k] < endXsponge) {
				factor = (((coordX[k]) / endXsponge) * c1o2) + c1o2;
				omega = factor * omegaIn;
			}
			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m2 = mfaaa + mfaac;
			m1 = mfaac - mfaaa;
			m0 = m2 + mfaab;
			mfaaa = m0;
			m0 += c1o36 * oMdrho;
			mfaab = m1 - m0 * vvz;
			mfaac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaba + mfabc;
			m1 = mfabc - mfaba;
			m0 = m2 + mfabb;
			mfaba = m0;
			m0 += c1o9 * oMdrho;
			mfabb = m1 - m0 * vvz;
			mfabc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaca + mfacc;
			m1 = mfacc - mfaca;
			m0 = m2 + mfacb;
			mfaca = m0;
			m0 += c1o36 * oMdrho;
			mfacb = m1 - m0 * vvz;
			mfacc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbaa + mfbac;
			m1 = mfbac - mfbaa;
			m0 = m2 + mfbab;
			mfbaa = m0;
			m0 += c1o9 * oMdrho;
			mfbab = m1 - m0 * vvz;
			mfbac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbba + mfbbc;
			m1 = mfbbc - mfbba;
			m0 = m2 + mfbbb;
			mfbba = m0;
			m0 += c4o9 * oMdrho;
			mfbbb = m1 - m0 * vvz;
			mfbbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbca + mfbcc;
			m1 = mfbcc - mfbca;
			m0 = m2 + mfbcb;
			mfbca = m0;
			m0 += c1o9 * oMdrho;
			mfbcb = m1 - m0 * vvz;
			mfbcc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcaa + mfcac;
			m1 = mfcac - mfcaa;
			m0 = m2 + mfcab;
			mfcaa = m0;
			m0 += c1o36 * oMdrho;
			mfcab = m1 - m0 * vvz;
			mfcac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcba + mfcbc;
			m1 = mfcbc - mfcba;
			m0 = m2 + mfcbb;
			mfcba = m0;
			m0 += c1o9 * oMdrho;
			mfcbb = m1 - m0 * vvz;
			mfcbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcca + mfccc;
			m1 = mfccc - mfcca;
			m0 = m2 + mfccb;
			mfcca = m0;
			m0 += c1o36 * oMdrho;
			mfccb = m1 - m0 * vvz;
			mfccc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2 = mfaaa + mfaca;
			m1 = mfaca - mfaaa;
			m0 = m2 + mfaba;
			mfaaa = m0;
			m0 += c1o6 * oMdrho;
			mfaba = m1 - m0 * vvy;
			mfaca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaab + mfacb;
			m1 = mfacb - mfaab;
			m0 = m2 + mfabb;
			mfaab = m0;
			mfabb = m1 - m0 * vvy;
			mfacb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaac + mfacc;
			m1 = mfacc - mfaac;
			m0 = m2 + mfabc;
			mfaac = m0;
			m0 += c1o18 * oMdrho;
			mfabc = m1 - m0 * vvy;
			mfacc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbaa + mfbca;
			m1 = mfbca - mfbaa;
			m0 = m2 + mfbba;
			mfbaa = m0;
			m0 += c2o3 * oMdrho;
			mfbba = m1 - m0 * vvy;
			mfbca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbab + mfbcb;
			m1 = mfbcb - mfbab;
			m0 = m2 + mfbbb;
			mfbab = m0;
			mfbbb = m1 - m0 * vvy;
			mfbcb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbac + mfbcc;
			m1 = mfbcc - mfbac;
			m0 = m2 + mfbbc;
			mfbac = m0;
			m0 += c2o9 * oMdrho;
			mfbbc = m1 - m0 * vvy;
			mfbcc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcaa + mfcca;
			m1 = mfcca - mfcaa;
			m0 = m2 + mfcba;
			mfcaa = m0;
			m0 += c1o6 * oMdrho;
			mfcba = m1 - m0 * vvy;
			mfcca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcab + mfccb;
			m1 = mfccb - mfcab;
			m0 = m2 + mfcbb;
			mfcab = m0;
			mfcbb = m1 - m0 * vvy;
			mfccb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcac + mfccc;
			m1 = mfccc - mfcac;
			m0 = m2 + mfcbc;
			mfcac = m0;
			m0 += c1o18 * oMdrho;
			mfcbc = m1 - m0 * vvy;
			mfccc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2 = mfaaa + mfcaa;
			m1 = mfcaa - mfaaa;
			m0 = m2 + mfbaa;
			mfaaa = m0;
			m0 += c1o1* oMdrho;
			mfbaa = m1 - m0 * vvx;
			mfcaa = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaba + mfcba;
			m1 = mfcba - mfaba;
			m0 = m2 + mfbba;
			mfaba = m0;
			mfbba = m1 - m0 * vvx;
			mfcba = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaca + mfcca;
			m1 = mfcca - mfaca;
			m0 = m2 + mfbca;
			mfaca = m0;
			m0 += c1o3 * oMdrho;
			mfbca = m1 - m0 * vvx;
			mfcca = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaab + mfcab;
			m1 = mfcab - mfaab;
			m0 = m2 + mfbab;
			mfaab = m0;
			mfbab = m1 - m0 * vvx;
			mfcab = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfabb + mfcbb;
			m1 = mfcbb - mfabb;
			m0 = m2 + mfbbb;
			mfabb = m0;
			mfbbb = m1 - m0 * vvx;
			mfcbb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfacb + mfccb;
			m1 = mfccb - mfacb;
			m0 = m2 + mfbcb;
			mfacb = m0;
			mfbcb = m1 - m0 * vvx;
			mfccb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaac + mfcac;
			m1 = mfcac - mfaac;
			m0 = m2 + mfbac;
			mfaac = m0;
			m0 += c1o3 * oMdrho;
			mfbac = m1 - m0 * vvx;
			mfcac = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfabc + mfcbc;
			m1 = mfcbc - mfabc;
			m0 = m2 + mfbbc;
			mfabc = m0;
			mfbbc = m1 - m0 * vvx;
			mfcbc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfacc + mfccc;
			m1 = mfccc - mfacc;
			m0 = m2 + mfbcc;
			mfacc = m0;
			m0 += c1o9 * oMdrho;
			mfbcc = m1 - m0 * vvx;
			mfccc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			real OxxPyyPzz = c1o1;

			////////////////////////////////////////////////////////////
			//3.
			//////////////////////////////
			real OxyyPxzz = c1o1;//two-omega;//(1000.*(-2. + omega))/(-1000. + 439.*omega);//(eight * (omega - two)) / (omega - eight);//two-omega;//eight*(two-omega)/(eight -omega);//one;//omega;//two-omega;//
			real OxyyMxzz = c1o1;//two-omega;//(1000.*(-2. + omega))/(-1000. + 439.*omega);//(eight * (omega - two)) / (omega - eight);//omega;//one;//eight*(two-omega)/(eight -omega);//one;//two-omega;//one;// 
								//////////////////////////////
								//real OxyyPxzz  = two-omega;//
								//real OxyyMxzz  = two-omega;//
								//////////////////////////////
								//real OxyyPxzz  = (eight * (omega - two)) / (omega - eight);//Ginzburg
								//real OxyyMxzz  = (eight * (omega - two)) / (omega - eight);//Ginzburg
								//////////////////////////////
								//real OxyyPxzz  = omega;//BGK
								//real OxyyMxzz  = omega;//BGK
								//////////////////////////////
								//real OxyyPxzz  = (one + omega) / two;//1P5
								//real OxyyMxzz  = (one + omega) / two;//1P5
								//////////////////////////////
								//real OxyyPxzz  = (three - omega) / two;//0P5
								//real OxyyMxzz  = (three - omega) / two;//0P5
								//////////////////////////////
								//real OxyyPxzz  = (one + (eight * (omega - two)) / (omega - eight)) / two;//one + Ginzburg / two ... Car
								//real OxyyMxzz  = (one + (eight * (omega - two)) / (omega - eight)) / two;//one + Ginzburg / two ... Car
								////////////////////////////////////////////////////////////
								//4.
								//////////////////////////////
			real O4 = c1o1;
			//////////////////////////////
			//real O4        = omega;//TRT
			////////////////////////////////////////////////////////////
			//5.
			//////////////////////////////
			real O5 = c1o1;
			////////////////////////////////////////////////////////////
			//6.
			//////////////////////////////
			real O6 = c1o1;
			////////////////////////////////////////////////////////////


			//central moments to cumulants
			//4.
			real CUMcbb = mfcbb - ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) / rho;
			real CUMbcb = mfbcb - ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) / rho;
			real CUMbbc = mfbbc - ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) / rho;

			real CUMcca = mfcca - (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho - c1o9*(drho / rho));
			real CUMcac = mfcac - (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho - c1o9*(drho / rho));
			real CUMacc = mfacc - (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho - c1o9*(drho / rho));

			//5.
			real CUMbcc = mfbcc - ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) / rho;
			real CUMcbc = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) / rho;
			real CUMccb = mfccb - ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) / rho;

			//6.

			real CUMccc = mfccc + ((-c4o1 *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
				+ (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
					+ c2o1 * (mfcaa * mfaca * mfaac)
					+ c16o1 *  mfbba * mfbab * mfabb) / (rho * rho)
				- c1o3 * (mfacc + mfcac + mfcca) / rho
				- c1o9 * (mfcaa + mfaca + mfaac) / rho
				+ (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
					+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3
				+ c1o27*((drho * drho - drho) / (rho*rho)));
			//+ c1o27*(one -three/rho +two/(rho*rho)));





			//2.
			// linear combinations
			real mxxPyyPzz = mfcaa + mfaca + mfaac;
			real mxxMyy = mfcaa - mfaca;
			real mxxMzz = mfcaa - mfaac;

			///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			//incl. correction		(hat noch nicht so gut funktioniert...Optimierungsbedarf??)
			{
				real dxux = c1o2 * (-omega) *(mxxMyy + mxxMzz) + c1o2 *  OxxPyyPzz * (mfaaa - mxxPyyPzz);
				real dyuy = dxux + omega * c3o2 * mxxMyy;
				real dzuz = dxux + omega * c3o2 * mxxMzz;

				//relax
				mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz) - c3o1 * (c1o1 - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2 * dzuz);//-magicBulk*OxxPyyPzz;
				mxxMyy += omega * (-mxxMyy) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy);
				mxxMzz += omega * (-mxxMzz) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz);

			}
			///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			////no correction
			//mxxPyyPzz += OxxPyyPzz*(mfaaa-mxxPyyPzz);//-magicBulk*OxxPyyPzz;
			//mxxMyy    += -(-omega) * (-mxxMyy);
			//mxxMzz    += -(-omega) * (-mxxMzz);
			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			mfabb += omega * (-mfabb);
			mfbab += omega * (-mfbab);
			mfbba += omega * (-mfbba);

			// linear combinations back
			mfcaa = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
			mfaca = c1o3 * (-c2o1*  mxxMyy + mxxMzz + mxxPyyPzz);
			mfaac = c1o3 * (mxxMyy - c2o1* mxxMzz + mxxPyyPzz);

			//3.
			// linear combinations

			real mxxyPyzz = mfcba + mfabc;
			real mxxyMyzz = mfcba - mfabc;

			real mxxzPyyz = mfcab + mfacb;
			real mxxzMyyz = mfcab - mfacb;

			real mxyyPxzz = mfbca + mfbac;
			real mxyyMxzz = mfbca - mfbac;

			//relax
			//////////////////////////////////////////////////////////////////////////
			//das ist der limiter
			wadjust = OxyyMxzz + (c1o1 - OxyyMxzz)*abs(mfbbb) / (abs(mfbbb) + qudricLimit);
			mfbbb += wadjust * (-mfbbb);
			wadjust = OxyyPxzz + (c1o1 - OxyyPxzz)*abs(mxxyPyzz) / (abs(mxxyPyzz) + qudricLimit);
			mxxyPyzz += wadjust * (-mxxyPyzz);
			wadjust = OxyyMxzz + (c1o1 - OxyyMxzz)*abs(mxxyMyzz) / (abs(mxxyMyzz) + qudricLimit);
			mxxyMyzz += wadjust * (-mxxyMyzz);
			wadjust = OxyyPxzz + (c1o1 - OxyyPxzz)*abs(mxxzPyyz) / (abs(mxxzPyyz) + qudricLimit);
			mxxzPyyz += wadjust * (-mxxzPyyz);
			wadjust = OxyyMxzz + (c1o1 - OxyyMxzz)*abs(mxxzMyyz) / (abs(mxxzMyyz) + qudricLimit);
			mxxzMyyz += wadjust * (-mxxzMyyz);
			wadjust = OxyyPxzz + (c1o1 - OxyyPxzz)*abs(mxyyPxzz) / (abs(mxyyPxzz) + qudricLimit);
			mxyyPxzz += wadjust * (-mxyyPxzz);
			wadjust = OxyyMxzz + (c1o1 - OxyyMxzz)*abs(mxyyMxzz) / (abs(mxyyMxzz) + qudricLimit);
			mxyyMxzz += wadjust * (-mxyyMxzz);
			//////////////////////////////////////////////////////////////////////////
			mfbbb += OxyyMxzz * (-mfbbb);
			mxxyPyzz += OxyyPxzz * (-mxxyPyzz);
			mxxyMyzz += OxyyMxzz * (-mxxyMyzz);
			mxxzPyyz += OxyyPxzz * (-mxxzPyyz);
			mxxzMyyz += OxyyMxzz * (-mxxzMyyz);
			mxyyPxzz += OxyyPxzz * (-mxyyPxzz);
			mxyyMxzz += OxyyMxzz * (-mxyyMxzz);

			//// linear combinations back

			mfcba = (mxxyMyzz + mxxyPyzz) * c1o2;
			mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
			mfcab = (mxxzMyyz + mxxzPyyz) * c1o2;
			mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
			mfbca = (mxyyMxzz + mxyyPxzz) * c1o2;
			mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;

			//4.
			//////////////////////////////////////////////////////////////////////////
			//mit limiter
			wadjust = O4 + (c1o1 - O4)*abs(CUMacc) / (abs(CUMacc) + qudricLimit);
			CUMacc += wadjust * (-CUMacc);
			wadjust = O4 + (c1o1 - O4)*abs(CUMcac) / (abs(CUMcac) + qudricLimit);
			CUMcac += wadjust * (-CUMcac);
			wadjust = O4 + (c1o1 - O4)*abs(CUMcca) / (abs(CUMcca) + qudricLimit);
			CUMcca += wadjust * (-CUMcca);

			wadjust = O4 + (c1o1 - O4)*abs(CUMbbc) / (abs(CUMbbc) + qudricLimit);
			CUMbbc += wadjust * (-CUMbbc);
			wadjust = O4 + (c1o1 - O4)*abs(CUMbcb) / (abs(CUMbcb) + qudricLimit);
			CUMbcb += wadjust * (-CUMbcb);
			wadjust = O4 + (c1o1 - O4)*abs(CUMcbb) / (abs(CUMcbb) + qudricLimit);
			CUMcbb += wadjust * (-CUMcbb);
			//////////////////////////////////////////////////////////////////////////
			//ohne limiter
			//CUMacc += O4 * (-CUMacc); 
			//CUMcac += O4 * (-CUMcac); 
			//CUMcca += O4 * (-CUMcca); 

			//CUMbbc += O4 * (-CUMbbc); 
			//CUMbcb += O4 * (-CUMbcb); 
			//CUMcbb += O4 * (-CUMcbb); 
			//////////////////////////////////////////////////////////////////////////


			//5.
			CUMbcc += O5 * (-CUMbcc);
			CUMcbc += O5 * (-CUMcbc);
			CUMccb += O5 * (-CUMccb);

			//6.
			CUMccc += O6 * (-CUMccc);



			//back cumulants to central moments
			//4.
			mfcbb = CUMcbb + ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) / rho;
			mfbcb = CUMbcb + ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) / rho;
			mfbbc = CUMbbc + ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) / rho;

			mfcca = CUMcca + (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho - c1o9*(drho / rho));
			mfcac = CUMcac + (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho - c1o9*(drho / rho));
			mfacc = CUMacc + (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho - c1o9*(drho / rho));

			//5.
			mfbcc = CUMbcc + ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) / rho;
			mfcbc = CUMcbc + ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) / rho;
			mfccb = CUMccb + ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) / rho;

			//6.

			mfccc = CUMccc - ((-c4o1 *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
				+ (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
					+ c2o1 * (mfcaa * mfaca * mfaac)
					+ c16o1 *  mfbba * mfbab * mfabb) / (rho * rho)
				- c1o3 * (mfacc + mfcac + mfcca) / rho
				- c1o9 * (mfcaa + mfaca + mfaac) / rho
				+ (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
					+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3
				+ c1o27*((drho * drho - drho) / (rho*rho)));
			//+ c1o27*(one -three/rho +two/(rho*rho)));


			////////////////////////////////////////////////////////////////////////////////////

			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			mfbaa = -mfbaa;
			mfaba = -mfaba;
			mfaab = -mfaab;
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			//back
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m0 = mfaac * c1o2 + mfaab * (vvz - c1o2) + (mfaaa + c1o1* oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfaac - c2o1* mfaab *  vvz + mfaaa                * (c1o1 - vz2) - c1o1* oMdrho * vz2;
			m2 = mfaac * c1o2 + mfaab * (vvz + c1o2) + (mfaaa + c1o1* oMdrho) * (vz2 + vvz) * c1o2;
			mfaaa = m0;
			mfaab = m1;
			mfaac = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfabc * c1o2 + mfabb * (vvz - c1o2) + mfaba * (vz2 - vvz) * c1o2;
			m1 = -mfabc - c2o1* mfabb *  vvz + mfaba * (c1o1 - vz2);
			m2 = mfabc * c1o2 + mfabb * (vvz + c1o2) + mfaba * (vz2 + vvz) * c1o2;
			mfaba = m0;
			mfabb = m1;
			mfabc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacc * c1o2 + mfacb * (vvz - c1o2) + (mfaca + c1o3 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfacc - c2o1* mfacb *  vvz + mfaca                  * (c1o1 - vz2) - c1o3 * oMdrho * vz2;
			m2 = mfacc * c1o2 + mfacb * (vvz + c1o2) + (mfaca + c1o3 * oMdrho) * (vz2 + vvz) * c1o2;
			mfaca = m0;
			mfacb = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfbac * c1o2 + mfbab * (vvz - c1o2) + mfbaa * (vz2 - vvz) * c1o2;
			m1 = -mfbac - c2o1* mfbab *  vvz + mfbaa * (c1o1 - vz2);
			m2 = mfbac * c1o2 + mfbab * (vvz + c1o2) + mfbaa * (vz2 + vvz) * c1o2;
			mfbaa = m0;
			mfbab = m1;
			mfbac = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbbc * c1o2 + mfbbb * (vvz - c1o2) + mfbba * (vz2 - vvz) * c1o2;
			m1 = -mfbbc - c2o1* mfbbb *  vvz + mfbba * (c1o1 - vz2);
			m2 = mfbbc * c1o2 + mfbbb * (vvz + c1o2) + mfbba * (vz2 + vvz) * c1o2;
			mfbba = m0;
			mfbbb = m1;
			mfbbc = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcc * c1o2 + mfbcb * (vvz - c1o2) + mfbca * (vz2 - vvz) * c1o2;
			m1 = -mfbcc - c2o1* mfbcb *  vvz + mfbca * (c1o1 - vz2);
			m2 = mfbcc * c1o2 + mfbcb * (vvz + c1o2) + mfbca * (vz2 + vvz) * c1o2;
			mfbca = m0;
			mfbcb = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcac * c1o2 + mfcab * (vvz - c1o2) + (mfcaa + c1o3 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfcac - c2o1* mfcab *  vvz + mfcaa                  * (c1o1 - vz2) - c1o3 * oMdrho * vz2;
			m2 = mfcac * c1o2 + mfcab * (vvz + c1o2) + (mfcaa + c1o3 * oMdrho) * (vz2 + vvz) * c1o2;
			mfcaa = m0;
			mfcab = m1;
			mfcac = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfcbc * c1o2 + mfcbb * (vvz - c1o2) + mfcba * (vz2 - vvz) * c1o2;
			m1 = -mfcbc - c2o1* mfcbb *  vvz + mfcba * (c1o1 - vz2);
			m2 = mfcbc * c1o2 + mfcbb * (vvz + c1o2) + mfcba * (vz2 + vvz) * c1o2;
			mfcba = m0;
			mfcbb = m1;
			mfcbc = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfccb * (vvz - c1o2) + (mfcca + c1o9 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfccc - c2o1* mfccb *  vvz + mfcca                  * (c1o1 - vz2) - c1o9 * oMdrho * vz2;
			m2 = mfccc * c1o2 + mfccb * (vvz + c1o2) + (mfcca + c1o9 * oMdrho) * (vz2 + vvz) * c1o2;
			mfcca = m0;
			mfccb = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m0 = mfaca * c1o2 + mfaba * (vvy - c1o2) + (mfaaa + c1o6 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfaca - c2o1* mfaba *  vvy + mfaaa                  * (c1o1 - vy2) - c1o6 * oMdrho * vy2;
			m2 = mfaca * c1o2 + mfaba * (vvy + c1o2) + (mfaaa + c1o6 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaaa = m0;
			mfaba = m1;
			mfaca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacb * c1o2 + mfabb * (vvy - c1o2) + (mfaab + c2o3 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfacb - c2o1* mfabb *  vvy + mfaab                  * (c1o1 - vy2) - c2o3 * oMdrho * vy2;
			m2 = mfacb * c1o2 + mfabb * (vvy + c1o2) + (mfaab + c2o3 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaab = m0;
			mfabb = m1;
			mfacb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacc * c1o2 + mfabc * (vvy - c1o2) + (mfaac + c1o6 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfacc - c2o1* mfabc *  vvy + mfaac                  * (c1o1 - vy2) - c1o6 * oMdrho * vy2;
			m2 = mfacc * c1o2 + mfabc * (vvy + c1o2) + (mfaac + c1o6 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaac = m0;
			mfabc = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfbca * c1o2 + mfbba * (vvy - c1o2) + mfbaa * (vy2 - vvy) * c1o2;
			m1 = -mfbca - c2o1* mfbba *  vvy + mfbaa * (c1o1 - vy2);
			m2 = mfbca * c1o2 + mfbba * (vvy + c1o2) + mfbaa * (vy2 + vvy) * c1o2;
			mfbaa = m0;
			mfbba = m1;
			mfbca = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcb * c1o2 + mfbbb * (vvy - c1o2) + mfbab * (vy2 - vvy) * c1o2;
			m1 = -mfbcb - c2o1* mfbbb *  vvy + mfbab * (c1o1 - vy2);
			m2 = mfbcb * c1o2 + mfbbb * (vvy + c1o2) + mfbab * (vy2 + vvy) * c1o2;
			mfbab = m0;
			mfbbb = m1;
			mfbcb = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcc * c1o2 + mfbbc * (vvy - c1o2) + mfbac * (vy2 - vvy) * c1o2;
			m1 = -mfbcc - c2o1* mfbbc *  vvy + mfbac * (c1o1 - vy2);
			m2 = mfbcc * c1o2 + mfbbc * (vvy + c1o2) + mfbac * (vy2 + vvy) * c1o2;
			mfbac = m0;
			mfbbc = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcca * c1o2 + mfcba * (vvy - c1o2) + (mfcaa + c1o18 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfcca - c2o1* mfcba *  vvy + mfcaa                   * (c1o1 - vy2) - c1o18 * oMdrho * vy2;
			m2 = mfcca * c1o2 + mfcba * (vvy + c1o2) + (mfcaa + c1o18 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcaa = m0;
			mfcba = m1;
			mfcca = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccb * c1o2 + mfcbb * (vvy - c1o2) + (mfcab + c2o9 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfccb - c2o1* mfcbb *  vvy + mfcab                  * (c1o1 - vy2) - c2o9 * oMdrho * vy2;
			m2 = mfccb * c1o2 + mfcbb * (vvy + c1o2) + (mfcab + c2o9 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcab = m0;
			mfcbb = m1;
			mfccb = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfcbc * (vvy - c1o2) + (mfcac + c1o18 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfccc - c2o1* mfcbc *  vvy + mfcac                   * (c1o1 - vy2) - c1o18 * oMdrho * vy2;
			m2 = mfccc * c1o2 + mfcbc * (vvy + c1o2) + (mfcac + c1o18 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcac = m0;
			mfcbc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m0 = mfcaa * c1o2 + mfbaa * (vvx - c1o2) + (mfaaa + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcaa - c2o1* mfbaa *  vvx + mfaaa                   * (c1o1 - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcaa * c1o2 + mfbaa * (vvx + c1o2) + (mfaaa + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaaa = m0;
			mfbaa = m1;
			mfcaa = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcba * c1o2 + mfbba * (vvx - c1o2) + (mfaba + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcba - c2o1* mfbba *  vvx + mfaba                  * (c1o1 - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcba * c1o2 + mfbba * (vvx + c1o2) + (mfaba + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaba = m0;
			mfbba = m1;
			mfcba = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcca * c1o2 + mfbca * (vvx - c1o2) + (mfaca + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcca - c2o1* mfbca *  vvx + mfaca                   * (c1o1 - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcca * c1o2 + mfbca * (vvx + c1o2) + (mfaca + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaca = m0;
			mfbca = m1;
			mfcca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcab * c1o2 + mfbab * (vvx - c1o2) + (mfaab + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcab - c2o1* mfbab *  vvx + mfaab                  * (c1o1 - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcab * c1o2 + mfbab * (vvx + c1o2) + (mfaab + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaab = m0;
			mfbab = m1;
			mfcab = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 = mfcbb * c1o2 + mfbbb * (vvx - c1o2) + (mfabb + c4o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcbb - c2o1* mfbbb *  vvx + mfabb                  * (c1o1 - vx2) - c4o9 * oMdrho * vx2;
			m2 = mfcbb * c1o2 + mfbbb * (vvx + c1o2) + (mfabb + c4o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfabb = m0;
			mfbbb = m1;
			mfcbb = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 = mfccb * c1o2 + mfbcb * (vvx - c1o2) + (mfacb + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfccb - c2o1* mfbcb *  vvx + mfacb                  * (c1o1 - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfccb * c1o2 + mfbcb * (vvx + c1o2) + (mfacb + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfacb = m0;
			mfbcb = m1;
			mfccb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcac * c1o2 + mfbac * (vvx - c1o2) + (mfaac + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcac - c2o1* mfbac *  vvx + mfaac                   * (c1o1 - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcac * c1o2 + mfbac * (vvx + c1o2) + (mfaac + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaac = m0;
			mfbac = m1;
			mfcac = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 = mfcbc * c1o2 + mfbbc * (vvx - c1o2) + (mfabc + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcbc - c2o1* mfbbc *  vvx + mfabc                  * (c1o1 - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcbc * c1o2 + mfbbc * (vvx + c1o2) + (mfabc + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfabc = m0;
			mfbbc = m1;
			mfcbc = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfbcc * (vvx - c1o2) + (mfacc + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfccc - c2o1* mfbcc *  vvx + mfacc                   * (c1o1 - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfccc * c1o2 + mfbcc * (vvx + c1o2) + (mfacc + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfacc = m0;
			mfbcc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////

			////////////////////////////////////////////////////////////////////////////////////
			(D.f[dirE])[k] = mfabb;
			(D.f[dirW])[kw] = mfcbb;
			(D.f[dirN])[k] = mfbab;
			(D.f[dirS])[ks] = mfbcb;
			(D.f[dirT])[k] = mfbba;
			(D.f[dirB])[kb] = mfbbc;
			(D.f[dirNE])[k] = mfaab;
			(D.f[dirSW])[ksw] = mfccb;
			(D.f[dirSE])[ks] = mfacb;
			(D.f[dirNW])[kw] = mfcab;
			(D.f[dirTE])[k] = mfaba;
			(D.f[dirBW])[kbw] = mfcbc;
			(D.f[dirBE])[kb] = mfabc;
			(D.f[dirTW])[kw] = mfcba;
			(D.f[dirTN])[k] = mfbaa;
			(D.f[dirBS])[kbs] = mfbcc;
			(D.f[dirBN])[kb] = mfbac;
			(D.f[dirTS])[ks] = mfbca;
			(D.f[dirZERO])[k] = mfbbb;
			(D.f[dirTNE])[k] = mfaaa;
			(D.f[dirTSE])[ks] = mfaca;
			(D.f[dirBNE])[kb] = mfaac;
			(D.f[dirBSE])[kbs] = mfacc;
			(D.f[dirTNW])[kw] = mfcaa;
			(D.f[dirTSW])[ksw] = mfcca;
			(D.f[dirBNW])[kbw] = mfcac;
			(D.f[dirBSW])[kbsw] = mfccc;
			////////////////////////////////////////////////////////////////////////////////////
		}
	}
}