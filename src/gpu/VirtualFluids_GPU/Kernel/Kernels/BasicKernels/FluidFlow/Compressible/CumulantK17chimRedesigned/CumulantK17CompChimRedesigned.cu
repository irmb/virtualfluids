#include "CumulantK17CompChimRedesigned.h"

#include "Parameter/Parameter.h"
#include "Parameter/CudaStreamManager.h"
#include "CumulantK17CompChimRedesigned_Device.cuh"

#include <hip/hip_runtime.h>

std::shared_ptr<CumulantK17CompChimRedesigned> CumulantK17CompChimRedesigned::getNewInstance(std::shared_ptr<Parameter> para,
                                                                               int level)
{
    return std::shared_ptr<CumulantK17CompChimRedesigned>(new CumulantK17CompChimRedesigned(para, level));
}

void CumulantK17CompChimRedesigned::run()
{
    LB_Kernel_CumulantK17CompChimRedesigned <<< cudaGrid.grid, cudaGrid.threads >>>(
        para->getParD(level)->omega,
        para->getParD(level)->neighborX,
        para->getParD(level)->neighborY,
        para->getParD(level)->neighborZ,
        para->getParD(level)->distributions.f[0],
        para->getParD(level)->numberOfNodes,
        level,
        para->getForcesDev(),
        para->getQuadricLimitersDev(),
        para->getParD(level)->rho,
        para->getParD(level)->velocityX,
        para->getParD(level)->velocityY,
        para->getParD(level)->velocityZ,
        para->getParD(level)->isEvenTimestep,
        para->getParD(level)->fluidNodeIndices,
        para->getParD(level)->numberOfFluidNodes);
    getLastCudaError("LB_Kernel_CumulantK17CompChim execution failed");
}

void CumulantK17CompChimRedesigned::runOnIndices(const unsigned int *indices, unsigned int size_indices, int streamIndex)
{
    hipStream_t stream = (streamIndex == -1) ? hipStreamLegacy : para->getStreamManager()->getStream(streamIndex);

    LB_Kernel_CumulantK17CompChimRedesigned<<< cudaGrid.grid, cudaGrid.threads, 0, stream>>>(
        para->getParD(level)->omega, 
        para->getParD(level)->neighborX, 
        para->getParD(level)->neighborY,
        para->getParD(level)->neighborZ, 
        para->getParD(level)->distributions.f[0], 
        para->getParD(level)->numberOfNodes, 
        level,
        para->getForcesDev(), 
        para->getQuadricLimitersDev(),
        para->getParD(level)->rho,
        para->getParD(level)->velocityX,
        para->getParD(level)->velocityY,
        para->getParD(level)->velocityZ,
        para->getParD(level)->isEvenTimestep,
        indices,
        size_indices);
    getLastCudaError("LB_Kernel_CumulantK17CompChim execution failed");
    
}

CumulantK17CompChimRedesigned::CumulantK17CompChimRedesigned(std::shared_ptr<Parameter> para, int level): KernelImp(para, level)
{
    myPreProcessorTypes.push_back(InitCompSP27);
    myKernelGroup = BasicKernel;
    this->cudaGrid = vf::cuda::CudaGrid(para->getParD(level)->numberofthreads, para->getParD(level)->numberOfNodes);
}

