#include "hip/hip_runtime.h"

// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file CumlantK17_Device.cu
//! \author Anna Wellmann, Martin Schönherr, Henry Korb, Henrik Asmuth
//! \date 05/12/2022
//! \brief Kernel for CumulantK17 including different turbulence models and options for local body forces and writing macroscopic variables
//!
//! CumulantK17 kernel using chimera transformations and quartic limiters as present in Geier et al. (2017). Additional options are three different
//! eddy-viscosity turbulence models (Smagorinsky, AMD, QR) that can be set via the template parameter turbulenceModel (with default
//! TurbulenceModel::None).
//! The kernel is executed separately for each subset of fluid node indices with a different tag CollisionTemplate. For each subset, only the locally
//! required options are switched on ( \param writeMacroscopicVariables and/or \param applyBodyForce) in order to minimize memory accesses. The default
//! refers to the plain cumlant kernel (CollisionTemplate::Default).
//! Nodes are added to subsets (taggedFluidNodes) in Simulation::init using a corresponding tag with different values of CollisionTemplate. These subsets
//! are provided by the utilized PostCollisionInteractiors depending on they specifc requirements (e.g. writeMacroscopicVariables for probes).

//=======================================================================================
/* Device code */
#include "LBM/LB.h"
#include "lbm/constants/D3Q27.h"
#include <lbm/constants/NumericConstants.h>
#include "Kernel/Utilities/DistributionHelper.cuh"

#include "GPU/TurbulentViscosityInlines.cuh"

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;
#include "Kernel/Utilities/ChimeraTransformation.h"


////////////////////////////////////////////////////////////////////////////////
template<TurbulenceModel turbulenceModel, bool writeMacroscopicVariables, bool applyBodyForce>
__global__ void LB_Kernel_CumulantK17(
    real omega_in,
    uint* neighborX,
    uint* neighborY,
    uint* neighborZ,
    real* distributions,
    real* rho,
    real* vx,
    real* vy,
    real* vz,
    real* turbulentViscosity,
    real SGSconstant,
    unsigned long long numberOfLBnodes,
    int level,
    real* forces,
    real* bodyForceX,
    real* bodyForceY,
    real* bodyForceZ,
    real* quadricLimiters,
    bool isEvenTimestep,
    const uint *fluidNodeIndices,
    uint numberOfFluidNodes)
{
    //////////////////////////////////////////////////////////////////////////
    //! Cumulant K17 Kernel is based on \ref
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040
    //! ]</b></a> and \ref <a href="https://doi.org/10.1016/j.jcp.2017.07.004"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.07.004 ]</b></a>
    //!
    //! The cumulant kernel is executed in the following steps
    //!
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned kThread = vf::gpu::getNodeIndex();

    //////////////////////////////////////////////////////////////////////////
    // run for all indices in size_Mat and fluid nodes
    if (kThread >= numberOfFluidNodes)
        return;
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get the node index from the array containing all indices of fluid nodes
    //!
    const unsigned k_000 = fluidNodeIndices[kThread];

    //////////////////////////////////////////////////////////////////////////
    //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on
    //! timestep is based on the esoteric twist algorithm \ref <a
    //! href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017),
    //! DOI:10.3390/computation5020019 ]</b></a>
    //!
    Distributions27 dist = vf::gpu::getDistributionReferences27(distributions, numberOfLBnodes, isEvenTimestep);

    ////////////////////////////////////////////////////////////////////////////////
    //! - Set neighbor indices (necessary for indirect addressing)
    uint k_M00 = neighborX[k_000];
    uint k_0M0 = neighborY[k_000];
    uint k_00M = neighborZ[k_000];
    uint k_MM0 = neighborY[k_M00];
    uint k_M0M = neighborZ[k_M00];
    uint k_0MM = neighborZ[k_0M0];
    uint k_MMM = neighborZ[k_MM0];
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Set local distributions
    //!
    real f_000 = (dist.f[DIR_000])[k_000];
    real f_P00 = (dist.f[DIR_P00])[k_000];
    real f_M00 = (dist.f[DIR_M00])[k_M00];
    real f_0P0 = (dist.f[DIR_0P0])[k_000];
    real f_0M0 = (dist.f[DIR_0M0])[k_0M0];
    real f_00P = (dist.f[DIR_00P])[k_000];
    real f_00M = (dist.f[DIR_00M])[k_00M];
    real f_PP0 = (dist.f[DIR_PP0])[k_000];
    real f_MM0 = (dist.f[DIR_MM0])[k_MM0];
    real f_PM0 = (dist.f[DIR_PM0])[k_0M0];
    real f_MP0 = (dist.f[DIR_MP0])[k_M00];
    real f_P0P = (dist.f[DIR_P0P])[k_000];
    real f_M0M = (dist.f[DIR_M0M])[k_M0M];
    real f_P0M = (dist.f[DIR_P0M])[k_00M];
    real f_M0P = (dist.f[DIR_M0P])[k_M00];
    real f_0PP = (dist.f[DIR_0PP])[k_000];
    real f_0MM = (dist.f[DIR_0MM])[k_0MM];
    real f_0PM = (dist.f[DIR_0PM])[k_00M];
    real f_0MP = (dist.f[DIR_0MP])[k_0M0];
    real f_PPP = (dist.f[DIR_PPP])[k_000];
    real f_MPP = (dist.f[DIR_MPP])[k_M00];
    real f_PMP = (dist.f[DIR_PMP])[k_0M0];
    real f_MMP = (dist.f[DIR_MMP])[k_MM0];
    real f_PPM = (dist.f[DIR_PPM])[k_00M];
    real f_MPM = (dist.f[DIR_MPM])[k_M0M];
    real f_PMM = (dist.f[DIR_PMM])[k_0MM];
    real f_MMM = (dist.f[DIR_MMM])[k_MMM];

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Define aliases to use the same variable for the moments (m's):
    //!
    real& m_111 = f_000;
    real& m_211 = f_P00;
    real& m_011 = f_M00;
    real& m_121 = f_0P0;
    real& m_101 = f_0M0;
    real& m_112 = f_00P;
    real& m_110 = f_00M;
    real& m_221 = f_PP0;
    real& m_001 = f_MM0;
    real& m_201 = f_PM0;
    real& m_021 = f_MP0;
    real& m_212 = f_P0P;
    real& m_010 = f_M0M;
    real& m_210 = f_P0M;
    real& m_012 = f_M0P;
    real& m_122 = f_0PP;
    real& m_100 = f_0MM;
    real& m_120 = f_0PM;
    real& m_102 = f_0MP;
    real& m_222 = f_PPP;
    real& m_022 = f_MPP;
    real& m_202 = f_PMP;
    real& m_002 = f_MMP;
    real& m_220 = f_PPM;
    real& m_020 = f_MPM;
    real& m_200 = f_PMM;
    real& m_000 = f_MMM;

    //////////////////////////////////////////////////////(unsigned long)//////////////////////////////
    //! - Calculate density and velocity using pyramid summation for low round-off errors as in Eq. (J1)-(J3) \ref
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
    //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a>
    //!
    real drho = ((((f_PPP + f_MMM) + (f_MPM + f_PMP)) + ((f_MPP + f_PMM) + (f_MMP + f_PPM))) +
                (((f_0MP + f_0PM) + (f_0MM + f_0PP)) + ((f_M0P + f_P0M) + (f_M0M + f_P0P)) +
                ((f_MP0 + f_PM0) + (f_MM0 + f_PP0))) +
                ((f_M00 + f_P00) + (f_0M0 + f_0P0) + (f_00M + f_00P))) +
                    f_000;

    real oneOverRho = c1o1 / (c1o1 + drho);

    real vvx = ((((f_PPP - f_MMM) + (f_PMP - f_MPM)) + ((f_PMM - f_MPP) + (f_PPM - f_MMP))) +
                (((f_P0M - f_M0P) + (f_P0P - f_M0M)) + ((f_PM0 - f_MP0) + (f_PP0 - f_MM0))) + (f_P00 - f_M00)) *
            oneOverRho;
    real vvy = ((((f_PPP - f_MMM) + (f_MPM - f_PMP)) + ((f_MPP - f_PMM) + (f_PPM - f_MMP))) +
                (((f_0PM - f_0MP) + (f_0PP - f_0MM)) + ((f_MP0 - f_PM0) + (f_PP0 - f_MM0))) + (f_0P0 - f_0M0)) *
            oneOverRho;
    real vvz = ((((f_PPP - f_MMM) + (f_PMP - f_MPM)) + ((f_MPP - f_PMM) + (f_MMP - f_PPM))) +
                (((f_0MP - f_0PM) + (f_0PP - f_0MM)) + ((f_M0P - f_P0M) + (f_P0P - f_M0M))) + (f_00P - f_00M)) *
            oneOverRho;

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Add half of the acceleration (body force) to the velocity as in Eq. (42) \ref
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
    //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a>
    //!
    real factor = c1o1;
    for (size_t i = 1; i <= level; i++) {
        factor *= c2o1;
    }

    real fx = forces[0];
    real fy = forces[1];
    real fz = forces[2];

    if( applyBodyForce ){
        fx += bodyForceX[k_000];
        fy += bodyForceY[k_000];
        fz += bodyForceZ[k_000];

        // real vx = vvx;
        // real vy = vvy;
        // real vz = vvz;
        real acc_x = fx * c1o2 / factor;
        real acc_y = fy * c1o2 / factor;
        real acc_z = fz * c1o2 / factor;

        vvx += acc_x;
        vvy += acc_y;
        vvz += acc_z;

        // Reset body force. To be used when not using round-off correction.
        bodyForceX[k_000] = 0.0f;
        bodyForceY[k_000] = 0.0f;
        bodyForceZ[k_000] = 0.0f;

        ////////////////////////////////////////////////////////////////////////////////////
        //!> Round-off correction
        //!
        //!> Similar to Kahan summation algorithm (https://en.wikipedia.org/wiki/Kahan_summation_algorithm)
        //!> Essentially computes the round-off error of the applied force and adds it in the next time step as a compensation.
        //!> Seems to be necesseary at very high Re boundary layers, where the forcing and velocity can
        //!> differ by several orders of magnitude.
        //!> \note 16/05/2022: Testing, still ongoing!
        //!
        // bodyForceX[k_000] = (acc_x-(vvx-vx))*factor*c2o1;
        // bodyForceY[k_000] = (acc_y-(vvy-vy))*factor*c2o1;
        // bodyForceZ[k_000] = (acc_z-(vvz-vz))*factor*c2o1;
    }
    else{
        vvx += fx * c1o2 / factor;
        vvy += fy * c1o2 / factor;
        vvz += fz * c1o2 / factor;
    }


    ////////////////////////////////////////////////////////////////////////////////////
    // calculate the square of velocities for this lattice node
    real vx2 = vvx * vvx;
    real vy2 = vvy * vvy;
    real vz2 = vvz * vvz;
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Set relaxation limiters for third order cumulants to default value \f$ \lambda=0.001 \f$ according to
    //! section 6 in \ref <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
    //!
    real quadricLimitP = quadricLimiters[0];
    real quadricLimitM = quadricLimiters[1];
    real quadricLimitD = quadricLimiters[2];
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Chimera transform from well conditioned distributions to central moments as defined in Appendix J in \ref
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
    //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a> see also Eq. (6)-(14) in \ref <a
    //! href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040
    //! ]</b></a>
    //!
    ////////////////////////////////////////////////////////////////////////////////////
    // Z - Dir
    forwardInverseChimeraWithK(f_MMM, f_MM0, f_MMP, vvz, vz2, c36o1, c1o36);
    forwardInverseChimeraWithK(f_M0M, f_M00, f_M0P, vvz, vz2, c9o1,  c1o9);
    forwardInverseChimeraWithK(f_MPM, f_MP0, f_MPP, vvz, vz2, c36o1, c1o36);
    forwardInverseChimeraWithK(f_0MM, f_0M0, f_0MP, vvz, vz2, c9o1,  c1o9);
    forwardInverseChimeraWithK(f_00M, f_000, f_00P, vvz, vz2, c9o4,  c4o9);
    forwardInverseChimeraWithK(f_0PM, f_0P0, f_0PP, vvz, vz2, c9o1,  c1o9);
    forwardInverseChimeraWithK(f_PMM, f_PM0, f_PMP, vvz, vz2, c36o1, c1o36);
    forwardInverseChimeraWithK(f_P0M, f_P00, f_P0P, vvz, vz2, c9o1,  c1o9);
    forwardInverseChimeraWithK(f_PPM, f_PP0, f_PPP, vvz, vz2, c36o1, c1o36);

    ////////////////////////////////////////////////////////////////////////////////////
    // Y - Dir
    forwardInverseChimeraWithK(f_MMM, f_M0M, f_MPM, vvy, vy2, c6o1,  c1o6);
    forwardChimera(            f_MM0, f_M00, f_MP0, vvy, vy2);
    forwardInverseChimeraWithK(f_MMP, f_M0P, f_MPP, vvy, vy2, c18o1, c1o18);
    forwardInverseChimeraWithK(f_0MM, f_00M, f_0PM, vvy, vy2, c3o2,  c2o3);
    forwardChimera(            f_0M0, f_000, f_0P0, vvy, vy2);
    forwardInverseChimeraWithK(f_0MP, f_00P, f_0PP, vvy, vy2, c9o2,  c2o9);
    forwardInverseChimeraWithK(f_PMM, f_P0M, f_PPM, vvy, vy2, c6o1,  c1o6);
    forwardChimera(            f_PM0, f_P00, f_PP0, vvy, vy2);
    forwardInverseChimeraWithK(f_PMP, f_P0P, f_PPP, vvy, vy2, c18o1, c1o18);

    ////////////////////////////////////////////////////////////////////////////////////
    // X - Dir
    forwardInverseChimeraWithK(f_MMM, f_0MM, f_PMM, vvx, vx2, c1o1, c1o1);
    forwardChimera(            f_M0M, f_00M, f_P0M, vvx, vx2);
    forwardInverseChimeraWithK(f_MPM, f_0PM, f_PPM, vvx, vx2, c3o1, c1o3);
    forwardChimera(            f_MM0, f_0M0, f_PM0, vvx, vx2);
    forwardChimera(            f_M00, f_000, f_P00, vvx, vx2);
    forwardChimera(            f_MP0, f_0P0, f_PP0, vvx, vx2);
    forwardInverseChimeraWithK(f_MMP, f_0MP, f_PMP, vvx, vx2, c3o1, c1o3);
    forwardChimera(            f_M0P, f_00P, f_P0P, vvx, vx2);
    forwardInverseChimeraWithK(f_MPP, f_0PP, f_PPP, vvx, vx2, c3o1, c1o9);

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Setting relaxation rates for non-hydrodynamic cumulants (default values). Variable names and equations
    //! according to <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
    //!  => [NAME IN PAPER]=[NAME IN CODE]=[DEFAULT VALUE].
    //!  - Trace of second order cumulants \f$ C_{200}+C_{020}+C_{002} \f$ used to adjust bulk
    //!  viscosity:\f$\omega_2=OxxPyyPzz=1.0 \f$.
    //!  - Third order cumulants \f$ C_{120}+C_{102}, C_{210}+C_{012}, C_{201}+C_{021} \f$: \f$ \omega_3=OxyyPxzz
    //!  \f$ set according to Eq. (111) with simplifications assuming \f$ \omega_2=1.0\f$.
    //!  - Third order cumulants \f$ C_{120}-C_{102}, C_{210}-C_{012}, C_{201}-C_{021} \f$: \f$ \omega_4 = OxyyMxzz
    //!  \f$ set according to Eq. (112) with simplifications assuming \f$ \omega_2 = 1.0\f$.
    //!  - Third order cumulants \f$ C_{111} \f$: \f$ \omega_5 = Oxyz \f$ set according to Eq. (113) with
    //!  simplifications assuming \f$ \omega_2 = 1.0\f$  (modify for different bulk viscosity).
    //!  - Fourth order cumulants \f$ C_{220}, C_{202}, C_{022}, C_{211}, C_{121}, C_{112} \f$: for simplification
    //!  all set to the same default value \f$ \omega_6=\omega_7=\omega_8=O4=1.0 \f$.
    //!  - Fifth order cumulants \f$ C_{221}, C_{212}, C_{122}\f$: \f$\omega_9=O5=1.0\f$.
    //!  - Sixth order cumulant \f$ C_{222}\f$: \f$\omega_{10}=O6=1.0\f$.
    //!
    ////////////////////////////////////////////////////////////////////////////////////
    //! - Calculate modified omega with turbulent viscosity
    //!
    real omega = omega_in;
    if(turbulenceModel != TurbulenceModel::None){ omega /= (c1o1 + c3o1*omega_in*turbulentViscosity[k_000]); }
    ////////////////////////////////////////////////////////////
    // 2.
    real OxxPyyPzz = c1o1;
    ////////////////////////////////////////////////////////////
    // 3.
    real OxyyPxzz = c8o1 * (-c2o1 + omega) * (c1o1 + c2o1 * omega) / (-c8o1 - c14o1 * omega + c7o1 * omega * omega);
    real OxyyMxzz =
        c8o1 * (-c2o1 + omega) * (-c7o1 + c4o1 * omega) / (c56o1 - c50o1 * omega + c9o1 * omega * omega);
    real Oxyz = c24o1 * (-c2o1 + omega) * (-c2o1 - c7o1 * omega + c3o1 * omega * omega) /
                (c48o1 + c152o1 * omega - c130o1 * omega * omega + c29o1 * omega * omega * omega);
    ////////////////////////////////////////////////////////////
    // 4.
    real O4 = c1o1;
    ////////////////////////////////////////////////////////////
    // 5.
    real O5 = c1o1;
    ////////////////////////////////////////////////////////////
    // 6.
    real O6 = c1o1;

    ////////////////////////////////////////////////////////////////////////////////////
    //! - A and DIR_00M: parameters for fourth order convergence of the diffusion term according to Eq. (115) and (116)
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a> with simplifications assuming \f$ \omega_2 = 1.0 \f$ (modify for
    //! different bulk viscosity).
    //!
    real factorA = (c4o1 + c2o1 * omega - c3o1 * omega * omega) / (c2o1 - c7o1 * omega + c5o1 * omega * omega);
    real factorB = (c4o1 + c28o1 * omega - c14o1 * omega * omega) / (c6o1 - c21o1 * omega + c15o1 * omega * omega);

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Compute cumulants from central moments according to Eq. (20)-(23) in
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
    //!
    ////////////////////////////////////////////////////////////
    // 4.
    real c_211 = m_211 - ((m_200 + c1o3) * m_011 + c2o1 * m_110 * m_101) * oneOverRho;
    real c_121 = m_121 - ((m_020 + c1o3) * m_101 + c2o1 * m_110 * m_011) * oneOverRho;
    real c_112 = m_112 - ((m_002 + c1o3) * m_110 + c2o1 * m_101 * m_011) * oneOverRho;

    real c_220 = m_220 - (((m_200 * m_020 + c2o1 * m_110 * m_110) + c1o3 * (m_200 + m_020)) * oneOverRho - c1o9 * (drho * oneOverRho));
    real c_202 = m_202 - (((m_200 * m_002 + c2o1 * m_101 * m_101) + c1o3 * (m_200 + m_002)) * oneOverRho - c1o9 * (drho * oneOverRho));
    real c_022 = m_022 - (((m_002 * m_020 + c2o1 * m_011 * m_011) + c1o3 * (m_002 + m_020)) * oneOverRho - c1o9 * (drho * oneOverRho));
    ////////////////////////////////////////////////////////////
    // 5.
    real c_122 =
        m_122 - ((m_002 * m_120 + m_020 * m_102 + c4o1 * m_011 * m_111 + c2o1 * (m_101 * m_021 + m_110 * m_012)) +
                c1o3 * (m_120 + m_102)) *
                oneOverRho;
    real c_212 =
        m_212 - ((m_002 * m_210 + m_200 * m_012 + c4o1 * m_101 * m_111 + c2o1 * (m_011 * m_201 + m_110 * m_102)) +
                c1o3 * (m_210 + m_012)) *
                oneOverRho;
    real c_221 =
        m_221 - ((m_200 * m_021 + m_020 * m_201 + c4o1 * m_110 * m_111 + c2o1 * (m_101 * m_120 + m_011 * m_210)) +
                c1o3 * (m_021 + m_201)) *
                oneOverRho;
    ////////////////////////////////////////////////////////////
    // 6.
    real c_222 = m_222 + ((-c4o1 * m_111 * m_111 - (m_200 * m_022 + m_020 * m_202 + m_002 * m_220) -
                            c4o1 * (m_011 * m_211 + m_101 * m_121 + m_110 * m_112) -
                            c2o1 * (m_120 * m_102 + m_210 * m_012 + m_201 * m_021)) *
                            oneOverRho +
                        (c4o1 * (m_101 * m_101 * m_020 + m_011 * m_011 * m_200 + m_110 * m_110 * m_002) +
                            c2o1 * (m_200 * m_020 * m_002) + c16o1 * m_110 * m_101 * m_011) *
                            oneOverRho * oneOverRho -
                            c1o3 * (m_022 + m_202 + m_220) * oneOverRho - c1o9 * (m_200 + m_020 + m_002) * oneOverRho +
                        (c2o1 * (m_101 * m_101 + m_011 * m_011 + m_110 * m_110) +
                            (m_002 * m_020 + m_002 * m_200 + m_020 * m_200) + c1o3 * (m_002 + m_020 + m_200)) *
                            oneOverRho * oneOverRho * c2o3 +
                            c1o27 * ((drho * drho - drho) * oneOverRho * oneOverRho));

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Compute linear combinations of second and third order cumulants
    //!
    ////////////////////////////////////////////////////////////
    // 2.
    real mxxPyyPzz = m_200 + m_020 + m_002;
    real mxxMyy    = m_200 - m_020;
    real mxxMzz    = m_200 - m_002;
    ////////////////////////////////////////////////////////////
    // 3.
    real mxxyPyzz = m_210 + m_012;
    real mxxyMyzz = m_210 - m_012;

    real mxxzPyyz = m_201 + m_021;
    real mxxzMyyz = m_201 - m_021;

    real mxyyPxzz = m_120 + m_102;
    real mxyyMxzz = m_120 - m_102;

    ////////////////////////////////////////////////////////////////////////////////////
    // incl. correction
    ////////////////////////////////////////////////////////////
    //! - Compute velocity  gradients from second order cumulants according to Eq. (27)-(32)
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a> Further explanations of the correction in viscosity in Appendix H of
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
    //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a> Note that the division by rho is omitted here as we need rho times
    //! the gradients later.
    //!
    real Dxy  = -c3o1 * omega * m_110;
    real Dxz  = -c3o1 * omega * m_101;
    real Dyz  = -c3o1 * omega * m_011;
    real dxux = c1o2 * (-omega) * (mxxMyy + mxxMzz) + c1o2 * OxxPyyPzz * (m_000 - mxxPyyPzz);
    real dyuy = dxux + omega * c3o2 * mxxMyy;
    real dzuz = dxux + omega * c3o2 * mxxMzz;

    ////////////////////////////////////////////////////////////////////////////////////
    switch (turbulenceModel)
    {
    case TurbulenceModel::None:
    case TurbulenceModel::AMD:  //AMD is computed in separate kernel
        break;
    case TurbulenceModel::Smagorinsky:
        turbulentViscosity[k_000] = calcTurbulentViscositySmagorinsky(SGSconstant, dxux, dyuy, dzuz, Dxy, Dxz , Dyz);
        break;
    case TurbulenceModel::QR:
        turbulentViscosity[k_000] = calcTurbulentViscosityQR(SGSconstant, dxux, dyuy, dzuz, Dxy, Dxz , Dyz);
        break;
    default:
        break;
    }
    ////////////////////////////////////////////////////////////
    //! - Relaxation of second order cumulants with correction terms according to Eq. (33)-(35) in
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
    //!
    mxxPyyPzz += OxxPyyPzz * (m_000 - mxxPyyPzz) - c3o1 * (c1o1 - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2 * dzuz);
    mxxMyy += omega * (-mxxMyy) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy);
    mxxMzz += omega * (-mxxMzz) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz);

    ////////////////////////////////////////////////////////////////////////////////////
    ////no correction
    // mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz);
    // mxxMyy += -(-omega) * (-mxxMyy);
    // mxxMzz += -(-omega) * (-mxxMzz);
    //////////////////////////////////////////////////////////////////////////
    m_011 += omega * (-m_011);
    m_101 += omega * (-m_101);
    m_110 += omega * (-m_110);

    ////////////////////////////////////////////////////////////////////////////////////
    // relax
    //////////////////////////////////////////////////////////////////////////
    // incl. limiter
    //! - Relaxation of third order cumulants including limiter according to Eq. (116)-(123)
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
    //!
    real wadjust = Oxyz + (c1o1 - Oxyz) * abs(m_111) / (abs(m_111) + quadricLimitD);
    m_111 += wadjust * (-m_111);
    wadjust = OxyyPxzz + (c1o1 - OxyyPxzz) * abs(mxxyPyzz) / (abs(mxxyPyzz) + quadricLimitP);
    mxxyPyzz += wadjust * (-mxxyPyzz);
    wadjust = OxyyMxzz + (c1o1 - OxyyMxzz) * abs(mxxyMyzz) / (abs(mxxyMyzz) + quadricLimitM);
    mxxyMyzz += wadjust * (-mxxyMyzz);
    wadjust = OxyyPxzz + (c1o1 - OxyyPxzz) * abs(mxxzPyyz) / (abs(mxxzPyyz) + quadricLimitP);
    mxxzPyyz += wadjust * (-mxxzPyyz);
    wadjust = OxyyMxzz + (c1o1 - OxyyMxzz) * abs(mxxzMyyz) / (abs(mxxzMyyz) + quadricLimitM);
    mxxzMyyz += wadjust * (-mxxzMyyz);
    wadjust = OxyyPxzz + (c1o1 - OxyyPxzz) * abs(mxyyPxzz) / (abs(mxyyPxzz) + quadricLimitP);
    mxyyPxzz += wadjust * (-mxyyPxzz);
    wadjust = OxyyMxzz + (c1o1 - OxyyMxzz) * abs(mxyyMxzz) / (abs(mxyyMxzz) + quadricLimitM);
    mxyyMxzz += wadjust * (-mxyyMxzz);
    //////////////////////////////////////////////////////////////////////////
    // no limiter
    // mfbbb += OxyyMxzz * (-mfbbb);
    // mxxyPyzz += OxyyPxzz * (-mxxyPyzz);
    // mxxyMyzz += OxyyMxzz * (-mxxyMyzz);
    // mxxzPyyz += OxyyPxzz * (-mxxzPyyz);
    // mxxzMyyz += OxyyMxzz * (-mxxzMyyz);
    // mxyyPxzz += OxyyPxzz * (-mxyyPxzz);
    // mxyyMxzz += OxyyMxzz * (-mxyyMxzz);

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Compute inverse linear combinations of second and third order cumulants
    //!
    m_200 = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
    m_020 = c1o3 * (-c2o1 * mxxMyy + mxxMzz + mxxPyyPzz);
    m_002 = c1o3 * (mxxMyy - c2o1 * mxxMzz + mxxPyyPzz);

    m_210 = ( mxxyMyzz + mxxyPyzz) * c1o2;
    m_012 = (-mxxyMyzz + mxxyPyzz) * c1o2;
    m_201 = ( mxxzMyyz + mxxzPyyz) * c1o2;
    m_021 = (-mxxzMyyz + mxxzPyyz) * c1o2;
    m_120 = ( mxyyMxzz + mxyyPxzz) * c1o2;
    m_102 = (-mxyyMxzz + mxyyPxzz) * c1o2;
    //////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////////////////
    // 4.
    // no limiter
    //! - Relax fourth order cumulants to modified equilibrium for fourth order convergence of diffusion according
    //! to Eq. (43)-(48) <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
    //!
    c_022 = -O4 * (c1o1 / omega - c1o2) * (dyuy + dzuz) * c2o3 * factorA + (c1o1 - O4) * (c_022);
    c_202 = -O4 * (c1o1 / omega - c1o2) * (dxux + dzuz) * c2o3 * factorA + (c1o1 - O4) * (c_202);
    c_220 = -O4 * (c1o1 / omega - c1o2) * (dyuy + dxux) * c2o3 * factorA + (c1o1 - O4) * (c_220);
    c_112 = -O4 * (c1o1 / omega - c1o2) * Dxy           * c1o3 * factorB + (c1o1 - O4) * (c_112);
    c_121 = -O4 * (c1o1 / omega - c1o2) * Dxz           * c1o3 * factorB + (c1o1 - O4) * (c_121);
    c_211 = -O4 * (c1o1 / omega - c1o2) * Dyz           * c1o3 * factorB + (c1o1 - O4) * (c_211);


    //////////////////////////////////////////////////////////////////////////
    // 5.
    c_122 += O5 * (-c_122);
    c_212 += O5 * (-c_212);
    c_221 += O5 * (-c_221);

    //////////////////////////////////////////////////////////////////////////
    // 6.
    c_222 += O6 * (-c_222);

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Compute central moments from post collision cumulants according to Eq. (53)-(56) in
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
    //!

    //////////////////////////////////////////////////////////////////////////
    // 4.
    m_211 = c_211 + c1o3 * ((c3o1 * m_200 + c1o1) * m_011 + c6o1 * m_110 * m_101) * oneOverRho;
    m_121 = c_121 + c1o3 * ((c3o1 * m_020 + c1o1) * m_101 + c6o1 * m_110 * m_011) * oneOverRho;
    m_112 = c_112 + c1o3 * ((c3o1 * m_002 + c1o1) * m_110 + c6o1 * m_101 * m_011) * oneOverRho;

    m_220 =
        c_220 + (((m_200 * m_020 + c2o1 * m_110 * m_110) * c9o1 + c3o1 * (m_200 + m_020)) * oneOverRho - (drho * oneOverRho)) * c1o9;
    m_202 =
        c_202 + (((m_200 * m_002 + c2o1 * m_101 * m_101) * c9o1 + c3o1 * (m_200 + m_002)) * oneOverRho - (drho * oneOverRho)) * c1o9;
    m_022 =
        c_022 + (((m_002 * m_020 + c2o1 * m_011 * m_011) * c9o1 + c3o1 * (m_002 + m_020)) * oneOverRho - (drho * oneOverRho)) * c1o9;

    //////////////////////////////////////////////////////////////////////////
    // 5.
    m_122 = c_122 + c1o3 *
            (c3o1 * (m_002 * m_120 + m_020 * m_102 + c4o1 * m_011 * m_111 + c2o1 * (m_101 * m_021 + m_110 * m_012)) +
            (m_120 + m_102)) * oneOverRho;
    m_212 = c_212 + c1o3 *
            (c3o1 * (m_002 * m_210 + m_200 * m_012 + c4o1 * m_101 * m_111 + c2o1 * (m_011 * m_201 + m_110 * m_102)) +
            (m_210 + m_012)) * oneOverRho;
    m_221 = c_221 + c1o3 *
            (c3o1 * (m_200 * m_021 + m_020 * m_201 + c4o1 * m_110 * m_111 + c2o1 * (m_101 * m_120 + m_011 * m_210)) +
            (m_021 + m_201)) * oneOverRho;

    //////////////////////////////////////////////////////////////////////////
    // 6.
    m_222 = c_222 - ((-c4o1 * m_111 * m_111 - (m_200 * m_022 + m_020 * m_202 + m_002 * m_220) -
                    c4o1 * (m_011 * m_211 + m_101 * m_121 + m_110 * m_112) -
                    c2o1 * (m_120 * m_102 + m_210 * m_012 + m_201 * m_021)) *
                    oneOverRho +
                    (c4o1 * (m_101 * m_101 * m_020 + m_011 * m_011 * m_200 + m_110 * m_110 * m_002) +
                    c2o1 * (m_200 * m_020 * m_002) + c16o1 * m_110 * m_101 * m_011) *
                    oneOverRho * oneOverRho -
                    c1o3 * (m_022 + m_202 + m_220) * oneOverRho - c1o9 * (m_200 + m_020 + m_002) * oneOverRho +
                    (c2o1 * (m_101 * m_101 + m_011 * m_011 + m_110 * m_110) +
                    (m_002 * m_020 + m_002 * m_200 + m_020 * m_200) + c1o3 * (m_002 + m_020 + m_200)) *
                    oneOverRho * oneOverRho * c2o3 +
                    c1o27 * ((drho * drho - drho) * oneOverRho * oneOverRho));

    ////////////////////////////////////////////////////////////////////////////////////
    //! -  Add acceleration (body force) to first order cumulants according to Eq. (85)-(87) in
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
    //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a>
    //!
    m_100 = -m_100;
    m_010 = -m_010;
    m_001 = -m_001;

    //Write to array here to distribute read/write
    if(writeMacroscopicVariables || turbulenceModel==TurbulenceModel::AMD)
    {
        rho[k_000] = drho;
        vx[k_000] = vvx;
        vy[k_000] = vvy;
        vz[k_000] = vvz;
    }

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Chimera transform from central moments to well conditioned distributions as defined in Appendix J in
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
    //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a> see also Eq. (88)-(96) in <a
    //! href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040
    //! ]</b></a>
    //!
    ////////////////////////////////////////////////////////////////////////////////////
    // X - Dir
    backwardInverseChimeraWithK(m_000, m_100, m_200, vvx, vx2, c1o1, c1o1);
    backwardChimera(            m_010, m_110, m_210, vvx, vx2);
    backwardInverseChimeraWithK(m_020, m_120, m_220, vvx, vx2, c3o1, c1o3);
    backwardChimera(            m_001, m_101, m_201, vvx, vx2);
    backwardChimera(            m_011, m_111, m_211, vvx, vx2);
    backwardChimera(            m_021, m_121, m_221, vvx, vx2);
    backwardInverseChimeraWithK(m_002, m_102, m_202, vvx, vx2, c3o1, c1o3);
    backwardChimera(            m_012, m_112, m_212, vvx, vx2);
    backwardInverseChimeraWithK(m_022, m_122, m_222, vvx, vx2, c9o1, c1o9);

    ////////////////////////////////////////////////////////////////////////////////////
    // Y - Dir
    backwardInverseChimeraWithK(m_000, m_010, m_020, vvy, vy2, c6o1, c1o6);
    backwardChimera(            m_001, m_011, m_021, vvy, vy2);
    backwardInverseChimeraWithK(m_002, m_012, m_022, vvy, vy2, c18o1, c1o18);
    backwardInverseChimeraWithK(m_100, m_110, m_120, vvy, vy2, c3o2, c2o3);
    backwardChimera(            m_101, m_111, m_121, vvy, vy2);
    backwardInverseChimeraWithK(m_102, m_112, m_122, vvy, vy2, c9o2, c2o9);
    backwardInverseChimeraWithK(m_200, m_210, m_220, vvy, vy2, c6o1, c1o6);
    backwardChimera(            m_201, m_211, m_221, vvy, vy2);
    backwardInverseChimeraWithK(m_202, m_212, m_222, vvy, vy2, c18o1, c1o18);

    ////////////////////////////////////////////////////////////////////////////////////
    // Z - Dir
    backwardInverseChimeraWithK(m_000, m_001, m_002, vvz, vz2, c36o1, c1o36);
    backwardInverseChimeraWithK(m_010, m_011, m_012, vvz, vz2, c9o1, c1o9);
    backwardInverseChimeraWithK(m_020, m_021, m_022, vvz, vz2, c36o1, c1o36);
    backwardInverseChimeraWithK(m_100, m_101, m_102, vvz, vz2, c9o1, c1o9);
    backwardInverseChimeraWithK(m_110, m_111, m_112, vvz, vz2, c9o4, c4o9);
    backwardInverseChimeraWithK(m_120, m_121, m_122, vvz, vz2, c9o1, c1o9);
    backwardInverseChimeraWithK(m_200, m_201, m_202, vvz, vz2, c36o1, c1o36);
    backwardInverseChimeraWithK(m_210, m_211, m_212, vvz, vz2, c9o1, c1o9);
    backwardInverseChimeraWithK(m_220, m_221, m_222, vvz, vz2, c36o1, c1o36);

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Write distributions: style of reading and writing the distributions from/to
    //! stored arrays dependent on timestep is based on the esoteric twist algorithm
    //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017),
    //! DOI:10.3390/computation5020019 ]</b></a>
    //!
    (dist.f[DIR_P00])[k_000] = f_M00;
    (dist.f[DIR_M00])[k_M00] = f_P00;
    (dist.f[DIR_0P0])[k_000] = f_0M0;
    (dist.f[DIR_0M0])[k_0M0] = f_0P0;
    (dist.f[DIR_00P])[k_000] = f_00M;
    (dist.f[DIR_00M])[k_00M] = f_00P;
    (dist.f[DIR_PP0])[k_000] = f_MM0;
    (dist.f[DIR_MM0])[k_MM0] = f_PP0;
    (dist.f[DIR_PM0])[k_0M0] = f_MP0;
    (dist.f[DIR_MP0])[k_M00] = f_PM0;
    (dist.f[DIR_P0P])[k_000] = f_M0M;
    (dist.f[DIR_M0M])[k_M0M] = f_P0P;
    (dist.f[DIR_P0M])[k_00M] = f_M0P;
    (dist.f[DIR_M0P])[k_M00] = f_P0M;
    (dist.f[DIR_0PP])[k_000] = f_0MM;
    (dist.f[DIR_0MM])[k_0MM] = f_0PP;
    (dist.f[DIR_0PM])[k_00M] = f_0MP;
    (dist.f[DIR_0MP])[k_0M0] = f_0PM;
    (dist.f[DIR_000])[k_000] = f_000;
    (dist.f[DIR_PPP])[k_000] = f_MMM;
    (dist.f[DIR_PMP])[k_0M0] = f_MPM;
    (dist.f[DIR_PPM])[k_00M] = f_MMP;
    (dist.f[DIR_PMM])[k_0MM] = f_MPP;
    (dist.f[DIR_MPP])[k_M00] = f_PMM;
    (dist.f[DIR_MMP])[k_MM0] = f_PPM;
    (dist.f[DIR_MPM])[k_M0M] = f_PMP;
    (dist.f[DIR_MMM])[k_MMM] = f_PPP;
}

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::AMD, true, true > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::Smagorinsky, true, true > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::QR, true, true > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::None, true, true > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::AMD, true, false > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::Smagorinsky, true, false > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::QR, true, false > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::None, true, false > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::AMD, false, true > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::Smagorinsky, false, true > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::QR, false, true > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::None, false, true > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::AMD, false, false > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::Smagorinsky, false, false > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::QR, false, false > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);

template __global__ void LB_Kernel_CumulantK17 < TurbulenceModel::None, false, false > ( real omega_in, uint* neighborX, uint* neighborY, uint* neighborZ, real* distributions, real* rho, real* vx, real* vy, real* vz, real* turbulentViscosity, real SGSconstant, unsigned long long numberOfLBnodes, int level, real* forces, real* bodyForceX, real* bodyForceY, real* bodyForceZ, real* quadricLimiters, bool isEvenTimestep, const uint *fluidNodeIndices, uint numberOfFluidNodes);
