#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __         
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |        
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |        
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |        
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____    
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|   
//      \    \  |    |   ________________________________________________________________    
//       \    \ |    |  |  ______________________________________________________________|   
//        \    \|    |  |  |         __          __     __     __     ______      _______    
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)   
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______    
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/   
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can 
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of 
//  the License, or (at your option) any later version.
//  
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT 
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or 
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License 
//  for more details.
//  
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file Cumulant27chim.cu
//! \ingroup GPU
//! \author Martin Schoenherr
//=======================================================================================
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;
#include "Kernel/ChimeraTransformation.h"

////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LB_Kernel_CumulantK17CompChimStream(
	real omega,
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	real* distributions,
	unsigned long size_Mat,
	int level,
	real* forces,
	real* quadricLimiters,
	bool isEvenTimestep,
    const uint *fluidNodeIndices, 
    uint numberOfFluidNodes)
{
    //////////////////////////////////////////////////////////////////////////
    //! Cumulant K17 Kernel is based on \ref
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040
    //! ]</b></a> and \ref <a href="https://doi.org/10.1016/j.jcp.2017.07.004"><b>[ M. Geier et al. (2017),
    //! DOI:10.1016/j.jcp.2017.07.004 ]</b></a>
    //!
    //! The cumulant kernel is executed in the following steps
    //!
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned x = threadIdx.x;
    const unsigned y = blockIdx.x;
    const unsigned z = blockIdx.y;

    const unsigned nx = blockDim.x;
    const unsigned ny = gridDim.x;

    const unsigned k_thread = nx * (ny * z + y) + x;

    //////////////////////////////////////////////////////////////////////////
    // run for all indices in fluidNodeIndices
    if (k_thread < numberOfFluidNodes) {
        //////////////////////////////////////////////////////////////////////////
        //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on
        //! timestep is based on the esoteric twist algorithm \ref <a
        //! href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017),
        //! DOI:10.3390/computation5020019 ]</b></a>

        const unsigned k = fluidNodeIndices[k_thread];

        Distributions27 dist;
        if (isEvenTimestep) {
            dist.f[dirE]    = &distributions[dirE * size_Mat];
            dist.f[dirW]    = &distributions[dirW * size_Mat];
            dist.f[dirN]    = &distributions[dirN * size_Mat];
            dist.f[dirS]    = &distributions[dirS * size_Mat];
            dist.f[dirT]    = &distributions[dirT * size_Mat];
            dist.f[dirB]    = &distributions[dirB * size_Mat];
            dist.f[dirNE]   = &distributions[dirNE * size_Mat];
            dist.f[dirSW]   = &distributions[dirSW * size_Mat];
            dist.f[dirSE]   = &distributions[dirSE * size_Mat];
            dist.f[dirNW]   = &distributions[dirNW * size_Mat];
            dist.f[dirTE]   = &distributions[dirTE * size_Mat];
            dist.f[dirBW]   = &distributions[dirBW * size_Mat];
            dist.f[dirBE]   = &distributions[dirBE * size_Mat];
            dist.f[dirTW]   = &distributions[dirTW * size_Mat];
            dist.f[dirTN]   = &distributions[dirTN * size_Mat];
            dist.f[dirBS]   = &distributions[dirBS * size_Mat];
            dist.f[dirBN]   = &distributions[dirBN * size_Mat];
            dist.f[dirTS]   = &distributions[dirTS * size_Mat];
            dist.f[dirZERO] = &distributions[dirZERO * size_Mat];
            dist.f[dirTNE]  = &distributions[dirTNE * size_Mat];
            dist.f[dirTSW]  = &distributions[dirTSW * size_Mat];
            dist.f[dirTSE]  = &distributions[dirTSE * size_Mat];
            dist.f[dirTNW]  = &distributions[dirTNW * size_Mat];
            dist.f[dirBNE]  = &distributions[dirBNE * size_Mat];
            dist.f[dirBSW]  = &distributions[dirBSW * size_Mat];
            dist.f[dirBSE]  = &distributions[dirBSE * size_Mat];
            dist.f[dirBNW]  = &distributions[dirBNW * size_Mat];
        } else {
            dist.f[dirW]    = &distributions[dirE * size_Mat];
            dist.f[dirE]    = &distributions[dirW * size_Mat];
            dist.f[dirS]    = &distributions[dirN * size_Mat];
            dist.f[dirN]    = &distributions[dirS * size_Mat];
            dist.f[dirB]    = &distributions[dirT * size_Mat];
            dist.f[dirT]    = &distributions[dirB * size_Mat];
            dist.f[dirSW]   = &distributions[dirNE * size_Mat];
            dist.f[dirNE]   = &distributions[dirSW * size_Mat];
            dist.f[dirNW]   = &distributions[dirSE * size_Mat];
            dist.f[dirSE]   = &distributions[dirNW * size_Mat];
            dist.f[dirBW]   = &distributions[dirTE * size_Mat];
            dist.f[dirTE]   = &distributions[dirBW * size_Mat];
            dist.f[dirTW]   = &distributions[dirBE * size_Mat];
            dist.f[dirBE]   = &distributions[dirTW * size_Mat];
            dist.f[dirBS]   = &distributions[dirTN * size_Mat];
            dist.f[dirTN]   = &distributions[dirBS * size_Mat];
            dist.f[dirTS]   = &distributions[dirBN * size_Mat];
            dist.f[dirBN]   = &distributions[dirTS * size_Mat];
            dist.f[dirZERO] = &distributions[dirZERO * size_Mat];
            dist.f[dirBSW]  = &distributions[dirTNE * size_Mat];
            dist.f[dirBNE]  = &distributions[dirTSW * size_Mat];
            dist.f[dirBNW]  = &distributions[dirTSE * size_Mat];
            dist.f[dirBSE]  = &distributions[dirTNW * size_Mat];
            dist.f[dirTSW]  = &distributions[dirBNE * size_Mat];
            dist.f[dirTNE]  = &distributions[dirBSW * size_Mat];
            dist.f[dirTNW]  = &distributions[dirBSE * size_Mat];
            dist.f[dirTSE]  = &distributions[dirBNW * size_Mat];
        }
        ////////////////////////////////////////////////////////////////////////////////
        //! - Set neighbor indices (necessary for indirect addressing)
        uint kw   = neighborX[k];
        uint ks   = neighborY[k];
        uint kb   = neighborZ[k];
        uint ksw  = neighborY[kw];
        uint kbw  = neighborZ[kw];
        uint kbs  = neighborZ[ks];
        uint kbsw = neighborZ[ksw];
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Set local distributions
        //!
        real mfcbb = (dist.f[dirE])[k];
        real mfabb = (dist.f[dirW])[kw];
        real mfbcb = (dist.f[dirN])[k];
        real mfbab = (dist.f[dirS])[ks];
        real mfbbc = (dist.f[dirT])[k];
        real mfbba = (dist.f[dirB])[kb];
        real mfccb = (dist.f[dirNE])[k];
        real mfaab = (dist.f[dirSW])[ksw];
        real mfcab = (dist.f[dirSE])[ks];
        real mfacb = (dist.f[dirNW])[kw];
        real mfcbc = (dist.f[dirTE])[k];
        real mfaba = (dist.f[dirBW])[kbw];
        real mfcba = (dist.f[dirBE])[kb];
        real mfabc = (dist.f[dirTW])[kw];
        real mfbcc = (dist.f[dirTN])[k];
        real mfbaa = (dist.f[dirBS])[kbs];
        real mfbca = (dist.f[dirBN])[kb];
        real mfbac = (dist.f[dirTS])[ks];
        real mfbbb = (dist.f[dirZERO])[k];
        real mfccc = (dist.f[dirTNE])[k];
        real mfaac = (dist.f[dirTSW])[ksw];
        real mfcac = (dist.f[dirTSE])[ks];
        real mfacc = (dist.f[dirTNW])[kw];
        real mfcca = (dist.f[dirBNE])[kb];
        real mfaaa = (dist.f[dirBSW])[kbsw];
        real mfcaa = (dist.f[dirBSE])[kbs];
        real mfaca = (dist.f[dirBNW])[kbw];
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Calculate density and velocity using pyramid summation for low round-off errors as in Eq. (J1)-(J3) \ref
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
        //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a>
        //!
        real drho = ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
                     (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) +
                      ((mfacb + mfcab) + (mfaab + mfccb))) +
                     ((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) +
                    mfbbb;

        real rho   = c1o1 + drho;
        real OOrho = c1o1 / rho;

        real vvx = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
                    (((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) + (mfcbb - mfabb)) *
                   OOrho;
        real vvy = ((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
                    (((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) + (mfbcb - mfbab)) *
                   OOrho;
        real vvz = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
                    (((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) + (mfbbc - mfbba)) *
                   OOrho;
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Add half of the acceleration (body force) to the velocity as in Eq. (42) \ref
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
        //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a>
        //!
        real factor = c1o1;
        for (size_t i = 1; i <= level; i++) {
            factor *= c2o1;
        }
        real fx = forces[0] / factor;
        real fy = forces[1] / factor;
        real fz = forces[2] / factor;
        vvx += fx * c1o2;
        vvy += fy * c1o2;
        vvz += fz * c1o2;
        ////////////////////////////////////////////////////////////////////////////////////
        // calculate the square of velocities for this lattice node
        real vx2 = vvx * vvx;
        real vy2 = vvy * vvy;
        real vz2 = vvz * vvz;
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Set relaxation limiters for third order cumulants to default value \f$ \lambda=0.001 \f$ according to
        //! section 6 in \ref <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
        //!
        real wadjust;
        real qudricLimitP = quadricLimiters[0];
        real qudricLimitM = quadricLimiters[1];
        real qudricLimitD = quadricLimiters[2];
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Chimera transform from well conditioned distributions to central moments as defined in Appendix J in \ref
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
        //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a> see also Eq. (6)-(14) in \ref <a
        //! href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040
        //! ]</b></a>
        //!
        ////////////////////////////////////////////////////////////////////////////////////
        // Z - Dir
        forwardInverseChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, c36o1, c1o36);
        forwardInverseChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2, c9o1, c1o9);
        forwardInverseChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c36o1, c1o36);
        forwardInverseChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2, c9o1, c1o9);
        forwardInverseChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2, c9o4, c4o9);
        forwardInverseChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2, c9o1, c1o9);
        forwardInverseChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c36o1, c1o36);
        forwardInverseChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2, c9o1, c1o9);
        forwardInverseChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c36o1, c1o36);

        ////////////////////////////////////////////////////////////////////////////////////
        // Y - Dir
        forwardInverseChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2, c6o1, c1o6);
        forwardChimera(mfaab, mfabb, mfacb, vvy, vy2);
        forwardInverseChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c18o1, c1o18);
        forwardInverseChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2, c3o2, c2o3);
        forwardChimera(mfbab, mfbbb, mfbcb, vvy, vy2);
        forwardInverseChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2, c9o2, c2o9);
        forwardInverseChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2, c6o1, c1o6);
        forwardChimera(mfcab, mfcbb, mfccb, vvy, vy2);
        forwardInverseChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c18o1, c1o18);

        ////////////////////////////////////////////////////////////////////////////////////
        // X - Dir
        forwardInverseChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o1, c1o1);
        forwardChimera(mfaba, mfbba, mfcba, vvx, vx2);
        forwardInverseChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c3o1, c1o3);
        forwardChimera(mfaab, mfbab, mfcab, vvx, vx2);
        forwardChimera(mfabb, mfbbb, mfcbb, vvx, vx2);
        forwardChimera(mfacb, mfbcb, mfccb, vvx, vx2);
        forwardInverseChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c3o1, c1o3);
        forwardChimera(mfabc, mfbbc, mfcbc, vvx, vx2);
        forwardInverseChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c3o1, c1o9);

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Setting relaxation rates for non-hydrodynamic cumulants (default values). Variable names and equations
        //! according to <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
        //!  => [NAME IN PAPER]=[NAME IN CODE]=[DEFAULT VALUE].
        //!  - Trace of second order cumulants \f$ C_{200}+C_{020}+C_{002} \f$ used to adjust bulk
        //!  viscosity:\f$\omega_2=OxxPyyPzz=1.0 \f$.
        //!  - Third order cumulants \f$ C_{120}+C_{102}, C_{210}+C_{012}, C_{201}+C_{021} \f$: \f$ \omega_3=OxyyPxzz
        //!  \f$ set according to Eq. (111) with simplifications assuming \f$ \omega_2=1.0\f$.
        //!  - Third order cumulants \f$ C_{120}-C_{102}, C_{210}-C_{012}, C_{201}-C_{021} \f$: \f$ \omega_4 = OxyyMxzz
        //!  \f$ set according to Eq. (112) with simplifications assuming \f$ \omega_2 = 1.0\f$.
        //!  - Third order cumulants \f$ C_{111} \f$: \f$ \omega_5 = Oxyz \f$ set according to Eq. (113) with
        //!  simplifications assuming \f$ \omega_2 = 1.0\f$  (modify for different bulk viscosity).
        //!  - Fourth order cumulants \f$ C_{220}, C_{202}, C_{022}, C_{211}, C_{121}, C_{112} \f$: for simplification
        //!  all set to the same default value \f$ \omega_6=\omega_7=\omega_8=O4=1.0 \f$.
        //!  - Fifth order cumulants \f$ C_{221}, C_{212}, C_{122}\f$: \f$\omega_9=O5=1.0\f$.
        //!  - Sixth order cumulant \f$ C_{222}\f$: \f$\omega_{10}=O6=1.0\f$.
        //!
        ////////////////////////////////////////////////////////////
        // 2.
        real OxxPyyPzz = c1o1;
        ////////////////////////////////////////////////////////////
        // 3.
        real OxyyPxzz = c8o1 * (-c2o1 + omega) * (c1o1 + c2o1 * omega) / (-c8o1 - c14o1 * omega + c7o1 * omega * omega);
        real OxyyMxzz =
            c8o1 * (-c2o1 + omega) * (-c7o1 + c4o1 * omega) / (c56o1 - c50o1 * omega + c9o1 * omega * omega);
        real Oxyz = c24o1 * (-c2o1 + omega) * (-c2o1 - c7o1 * omega + c3o1 * omega * omega) /
                    (c48o1 + c152o1 * omega - c130o1 * omega * omega + c29o1 * omega * omega * omega);
        ////////////////////////////////////////////////////////////
        // 4.
        real O4 = c1o1;
        ////////////////////////////////////////////////////////////
        // 5.
        real O5 = c1o1;
        ////////////////////////////////////////////////////////////
        // 6.
        real O6 = c1o1;

        ////////////////////////////////////////////////////////////////////////////////////
        //! - A and B: parameters for fourth order convergence of the diffusion term according to Eq. (114) and (115)
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a> with simplifications assuming \f$ \omega_2 = 1.0 \f$ (modify for
        //! different bulk viscosity).
        //!
        real A = (c4o1 + c2o1 * omega - c3o1 * omega * omega) / (c2o1 - c7o1 * omega + c5o1 * omega * omega);
        real B = (c4o1 + c28o1 * omega - c14o1 * omega * omega) / (c6o1 - c21o1 * omega + c15o1 * omega * omega);

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Compute cumulants from central moments according to Eq. (20)-(23) in
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
        //!
        ////////////////////////////////////////////////////////////
        // 4.
        real CUMcbb = mfcbb - ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) * OOrho;
        real CUMbcb = mfbcb - ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) * OOrho;
        real CUMbbc = mfbbc - ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) * OOrho;

        real CUMcca =
            mfcca - (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) * OOrho - c1o9 * (drho * OOrho));
        real CUMcac =
            mfcac - (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) * OOrho - c1o9 * (drho * OOrho));
        real CUMacc =
            mfacc - (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) * OOrho - c1o9 * (drho * OOrho));
        ////////////////////////////////////////////////////////////
        // 5.
        real CUMbcc =
            mfbcc - ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) +
                     c1o3 * (mfbca + mfbac)) *
                        OOrho;
        real CUMcbc =
            mfcbc - ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) +
                     c1o3 * (mfcba + mfabc)) *
                        OOrho;
        real CUMccb =
            mfccb - ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) +
                     c1o3 * (mfacb + mfcab)) *
                        OOrho;
        ////////////////////////////////////////////////////////////
        // 6.
        real CUMccc = mfccc + ((-c4o1 * mfbbb * mfbbb - (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca) -
                                c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc) -
                                c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) *
                                   OOrho +
                               (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac) +
                                c2o1 * (mfcaa * mfaca * mfaac) + c16o1 * mfbba * mfbab * mfabb) *
                                   OOrho * OOrho -
                               c1o3 * (mfacc + mfcac + mfcca) * OOrho - c1o9 * (mfcaa + mfaca + mfaac) * OOrho +
                               (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba) +
                                (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 * (mfaac + mfaca + mfcaa)) *
                                   OOrho * OOrho * c2o3 +
                               c1o27 * ((drho * drho - drho) * OOrho * OOrho));

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Compute linear combinations of second and third order cumulants
        //!
        ////////////////////////////////////////////////////////////
        // 2.
        real mxxPyyPzz = mfcaa + mfaca + mfaac;
        real mxxMyy    = mfcaa - mfaca;
        real mxxMzz    = mfcaa - mfaac;
        ////////////////////////////////////////////////////////////
        // 3.
        real mxxyPyzz = mfcba + mfabc;
        real mxxyMyzz = mfcba - mfabc;

        real mxxzPyyz = mfcab + mfacb;
        real mxxzMyyz = mfcab - mfacb;

        real mxyyPxzz = mfbca + mfbac;
        real mxyyMxzz = mfbca - mfbac;

        ////////////////////////////////////////////////////////////////////////////////////
        // incl. correction
        ////////////////////////////////////////////////////////////
        //! - Compute velocity  gradients from second order cumulants according to Eq. (27)-(32)
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a> Further explanations of the correction in viscosity in Appendix H of
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
        //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a> Note that the division by rho is omitted here as we need rho times
        //! the gradients later.
        //!
        real Dxy  = -c3o1 * omega * mfbba;
        real Dxz  = -c3o1 * omega * mfbab;
        real Dyz  = -c3o1 * omega * mfabb;
        real dxux = c1o2 * (-omega) * (mxxMyy + mxxMzz) + c1o2 * OxxPyyPzz * (mfaaa - mxxPyyPzz);
        real dyuy = dxux + omega * c3o2 * mxxMyy;
        real dzuz = dxux + omega * c3o2 * mxxMzz;
        ////////////////////////////////////////////////////////////
        //! - Relaxation of second order cumulants with correction terms according to Eq. (33)-(35) in
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
        //!
        mxxPyyPzz +=
            OxxPyyPzz * (mfaaa - mxxPyyPzz) - c3o1 * (c1o1 - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2 * dzuz);
        mxxMyy += omega * (-mxxMyy) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy);
        mxxMzz += omega * (-mxxMzz) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz);

        ////////////////////////////////////////////////////////////////////////////////////
        ////no correction
        // mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz);
        // mxxMyy += -(-omega) * (-mxxMyy);
        // mxxMzz += -(-omega) * (-mxxMzz);
        //////////////////////////////////////////////////////////////////////////
        mfabb += omega * (-mfabb);
        mfbab += omega * (-mfbab);
        mfbba += omega * (-mfbba);

        ////////////////////////////////////////////////////////////////////////////////////
        // relax
        //////////////////////////////////////////////////////////////////////////
        // incl. limiter
        //! - Relaxation of third order cumulants including limiter according to Eq. (116)-(123)
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
        //!
        wadjust = Oxyz + (c1o1 - Oxyz) * abs(mfbbb) / (abs(mfbbb) + qudricLimitD);
        mfbbb += wadjust * (-mfbbb);
        wadjust = OxyyPxzz + (c1o1 - OxyyPxzz) * abs(mxxyPyzz) / (abs(mxxyPyzz) + qudricLimitP);
        mxxyPyzz += wadjust * (-mxxyPyzz);
        wadjust = OxyyMxzz + (c1o1 - OxyyMxzz) * abs(mxxyMyzz) / (abs(mxxyMyzz) + qudricLimitM);
        mxxyMyzz += wadjust * (-mxxyMyzz);
        wadjust = OxyyPxzz + (c1o1 - OxyyPxzz) * abs(mxxzPyyz) / (abs(mxxzPyyz) + qudricLimitP);
        mxxzPyyz += wadjust * (-mxxzPyyz);
        wadjust = OxyyMxzz + (c1o1 - OxyyMxzz) * abs(mxxzMyyz) / (abs(mxxzMyyz) + qudricLimitM);
        mxxzMyyz += wadjust * (-mxxzMyyz);
        wadjust = OxyyPxzz + (c1o1 - OxyyPxzz) * abs(mxyyPxzz) / (abs(mxyyPxzz) + qudricLimitP);
        mxyyPxzz += wadjust * (-mxyyPxzz);
        wadjust = OxyyMxzz + (c1o1 - OxyyMxzz) * abs(mxyyMxzz) / (abs(mxyyMxzz) + qudricLimitM);
        mxyyMxzz += wadjust * (-mxyyMxzz);
        //////////////////////////////////////////////////////////////////////////
        // no limiter
        // mfbbb += OxyyMxzz * (-mfbbb);
        // mxxyPyzz += OxyyPxzz * (-mxxyPyzz);
        // mxxyMyzz += OxyyMxzz * (-mxxyMyzz);
        // mxxzPyyz += OxyyPxzz * (-mxxzPyyz);
        // mxxzMyyz += OxyyMxzz * (-mxxzMyyz);
        // mxyyPxzz += OxyyPxzz * (-mxyyPxzz);
        // mxyyMxzz += OxyyMxzz * (-mxyyMxzz);

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Compute inverse linear combinations of second and third order cumulants
        //!
        mfcaa = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
        mfaca = c1o3 * (-c2o1 * mxxMyy + mxxMzz + mxxPyyPzz);
        mfaac = c1o3 * (mxxMyy - c2o1 * mxxMzz + mxxPyyPzz);

        mfcba = (mxxyMyzz + mxxyPyzz) * c1o2;
        mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
        mfcab = (mxxzMyyz + mxxzPyyz) * c1o2;
        mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
        mfbca = (mxyyMxzz + mxyyPxzz) * c1o2;
        mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;
        //////////////////////////////////////////////////////////////////////////

        //////////////////////////////////////////////////////////////////////////
        // 4.
        // no limiter
        //! - Relax fourth order cumulants to modified equilibrium for fourth order convergence of diffusion according
        //! to Eq. (43)-(48) <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
        //!
        CUMacc = -O4 * (c1o1 / omega - c1o2) * (dyuy + dzuz) * c2o3 * A + (c1o1 - O4) * (CUMacc);
        CUMcac = -O4 * (c1o1 / omega - c1o2) * (dxux + dzuz) * c2o3 * A + (c1o1 - O4) * (CUMcac);
        CUMcca = -O4 * (c1o1 / omega - c1o2) * (dyuy + dxux) * c2o3 * A + (c1o1 - O4) * (CUMcca);
        CUMbbc = -O4 * (c1o1 / omega - c1o2) * Dxy * c1o3 * B + (c1o1 - O4) * (CUMbbc);
        CUMbcb = -O4 * (c1o1 / omega - c1o2) * Dxz * c1o3 * B + (c1o1 - O4) * (CUMbcb);
        CUMcbb = -O4 * (c1o1 / omega - c1o2) * Dyz * c1o3 * B + (c1o1 - O4) * (CUMcbb);

        //////////////////////////////////////////////////////////////////////////
        // 5.
        CUMbcc += O5 * (-CUMbcc);
        CUMcbc += O5 * (-CUMcbc);
        CUMccb += O5 * (-CUMccb);

        //////////////////////////////////////////////////////////////////////////
        // 6.
        CUMccc += O6 * (-CUMccc);

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Compute central moments from post collision cumulants according to Eq. (53)-(56) in
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017),
        //! DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
        //!

        //////////////////////////////////////////////////////////////////////////
        // 4.
        mfcbb = CUMcbb + c1o3 * ((c3o1 * mfcaa + c1o1) * mfabb + c6o1 * mfbba * mfbab) * OOrho;
        mfbcb = CUMbcb + c1o3 * ((c3o1 * mfaca + c1o1) * mfbab + c6o1 * mfbba * mfabb) * OOrho;
        mfbbc = CUMbbc + c1o3 * ((c3o1 * mfaac + c1o1) * mfbba + c6o1 * mfbab * mfabb) * OOrho;

        mfcca =
            CUMcca +
            (((mfcaa * mfaca + c2o1 * mfbba * mfbba) * c9o1 + c3o1 * (mfcaa + mfaca)) * OOrho - (drho * OOrho)) * c1o9;
        mfcac =
            CUMcac +
            (((mfcaa * mfaac + c2o1 * mfbab * mfbab) * c9o1 + c3o1 * (mfcaa + mfaac)) * OOrho - (drho * OOrho)) * c1o9;
        mfacc =
            CUMacc +
            (((mfaac * mfaca + c2o1 * mfabb * mfabb) * c9o1 + c3o1 * (mfaac + mfaca)) * OOrho - (drho * OOrho)) * c1o9;

        //////////////////////////////////////////////////////////////////////////
        // 5.
        mfbcc = CUMbcc + c1o3 *
                             (c3o1 * (mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb +
                                      c2o1 * (mfbab * mfacb + mfbba * mfabc)) +
                              (mfbca + mfbac)) *
                             OOrho;
        mfcbc = CUMcbc + c1o3 *
                             (c3o1 * (mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb +
                                      c2o1 * (mfabb * mfcab + mfbba * mfbac)) +
                              (mfcba + mfabc)) *
                             OOrho;
        mfccb = CUMccb + c1o3 *
                             (c3o1 * (mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb +
                                      c2o1 * (mfbab * mfbca + mfabb * mfcba)) +
                              (mfacb + mfcab)) *
                             OOrho;

        //////////////////////////////////////////////////////////////////////////
        // 6.
        mfccc = CUMccc - ((-c4o1 * mfbbb * mfbbb - (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca) -
                           c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc) -
                           c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) *
                              OOrho +
                          (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac) +
                           c2o1 * (mfcaa * mfaca * mfaac) + c16o1 * mfbba * mfbab * mfabb) *
                              OOrho * OOrho -
                          c1o3 * (mfacc + mfcac + mfcca) * OOrho - c1o9 * (mfcaa + mfaca + mfaac) * OOrho +
                          (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba) +
                           (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 * (mfaac + mfaca + mfcaa)) *
                              OOrho * OOrho * c2o3 +
                          c1o27 * ((drho * drho - drho) * OOrho * OOrho));

        ////////////////////////////////////////////////////////////////////////////////////
        //! -  Add acceleration (body force) to first order cumulants according to Eq. (85)-(87) in
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
        //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a>
        //!
        mfbaa = -mfbaa;
        mfaba = -mfaba;
        mfaab = -mfaab;

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Chimera transform from central moments to well conditioned distributions as defined in Appendix J in
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
        //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a> see also Eq. (88)-(96) in <a
        //! href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040
        //! ]</b></a>
        //!
        ////////////////////////////////////////////////////////////////////////////////////
        // X - Dir
        backwardInverseChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o1, c1o1);
        backwardChimera(mfaba, mfbba, mfcba, vvx, vx2);
        backwardInverseChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c3o1, c1o3);
        backwardChimera(mfaab, mfbab, mfcab, vvx, vx2);
        backwardChimera(mfabb, mfbbb, mfcbb, vvx, vx2);
        backwardChimera(mfacb, mfbcb, mfccb, vvx, vx2);
        backwardInverseChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c3o1, c1o3);
        backwardChimera(mfabc, mfbbc, mfcbc, vvx, vx2);
        backwardInverseChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c9o1, c1o9);

        ////////////////////////////////////////////////////////////////////////////////////
        // Y - Dir
        backwardInverseChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2, c6o1, c1o6);
        backwardChimera(mfaab, mfabb, mfacb, vvy, vy2);
        backwardInverseChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c18o1, c1o18);
        backwardInverseChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2, c3o2, c2o3);
        backwardChimera(mfbab, mfbbb, mfbcb, vvy, vy2);
        backwardInverseChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2, c9o2, c2o9);
        backwardInverseChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2, c6o1, c1o6);
        backwardChimera(mfcab, mfcbb, mfccb, vvy, vy2);
        backwardInverseChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c18o1, c1o18);

        ////////////////////////////////////////////////////////////////////////////////////
        // Z - Dir
        backwardInverseChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, c36o1, c1o36);
        backwardInverseChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2, c9o1, c1o9);
        backwardInverseChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c36o1, c1o36);
        backwardInverseChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2, c9o1, c1o9);
        backwardInverseChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2, c9o4, c4o9);
        backwardInverseChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2, c9o1, c1o9);
        backwardInverseChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c36o1, c1o36);
        backwardInverseChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2, c9o1, c1o9);
        backwardInverseChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c36o1, c1o36);

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Write distributions: style of reading and writing the distributions from/to
        //! stored arrays dependent on timestep is based on the esoteric twist algorithm
        //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017),
        //! DOI:10.3390/computation5020019 ]</b></a>
        //!
        (dist.f[dirE])[k]      = mfabb;
        (dist.f[dirW])[kw]     = mfcbb;
        (dist.f[dirN])[k]      = mfbab;
        (dist.f[dirS])[ks]     = mfbcb;
        (dist.f[dirT])[k]      = mfbba;
        (dist.f[dirB])[kb]     = mfbbc;
        (dist.f[dirNE])[k]     = mfaab;
        (dist.f[dirSW])[ksw]   = mfccb;
        (dist.f[dirSE])[ks]    = mfacb;
        (dist.f[dirNW])[kw]    = mfcab;
        (dist.f[dirTE])[k]     = mfaba;
        (dist.f[dirBW])[kbw]   = mfcbc;
        (dist.f[dirBE])[kb]    = mfabc;
        (dist.f[dirTW])[kw]    = mfcba;
        (dist.f[dirTN])[k]     = mfbaa;
        (dist.f[dirBS])[kbs]   = mfbcc;
        (dist.f[dirBN])[kb]    = mfbac;
        (dist.f[dirTS])[ks]    = mfbca;
        (dist.f[dirZERO])[k]   = mfbbb;
        (dist.f[dirTNE])[k]    = mfaaa;
        (dist.f[dirTSE])[ks]   = mfaca;
        (dist.f[dirBNE])[kb]   = mfaac;
        (dist.f[dirBSE])[kbs]  = mfacc;
        (dist.f[dirTNW])[kw]   = mfcaa;
        (dist.f[dirTSW])[ksw]  = mfcca;
        (dist.f[dirBNW])[kbw]  = mfcac;
        (dist.f[dirBSW])[kbsw] = mfccc;
    }
}