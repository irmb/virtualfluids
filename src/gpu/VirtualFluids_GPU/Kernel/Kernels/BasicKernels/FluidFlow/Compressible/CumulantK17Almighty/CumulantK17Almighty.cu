#include "CumulantK17Almighty.h"
#include <logger/Logger.h>
#include "Parameter/Parameter.h"
#include "Parameter/CudaStreamManager.h"
#include "CumulantK17Almighty_Device.cuh"

#include <hip/hip_runtime.h>

template<TurbulenceModel turbulenceModel> 
std::shared_ptr< CumulantK17Almighty<turbulenceModel> > CumulantK17Almighty<turbulenceModel>::getNewInstance(std::shared_ptr<Parameter> para, int level)
{
	return std::shared_ptr<CumulantK17Almighty<turbulenceModel> >(new CumulantK17Almighty<turbulenceModel>(para,level));
}

template<TurbulenceModel turbulenceModel>
void CumulantK17Almighty<turbulenceModel>::run()
{
	LB_Kernel_CumulantK17Almighty < turbulenceModel, false, false  > <<< cudaGrid.grid, cudaGrid.threads >>>(   para->getParD(level)->omega, 	
																												para->getParD(level)->typeOfGridNode, 										
																												para->getParD(level)->neighborX, para->getParD(level)->neighborY, para->getParD(level)->neighborZ,	
																												para->getParD(level)->distributions.f[0],	
																												para->getParD(level)->rho,		
																												para->getParD(level)->velocityX, para->getParD(level)->velocityY, para->getParD(level)->velocityZ,	
																												para->getParD(level)->turbViscosity,
																												para->getSGSConstant(),
																												(unsigned long)para->getParD(level)->numberOfNodes,	
																												level,				
																												para->getIsBodyForce(),				
																												para->getForcesDev(),				
																												para->getParD(level)->forceX_SP, para->getParD(level)->forceY_SP, para->getParD(level)->forceZ_SP,
																												para->getQuadricLimitersDev(),			
																												para->getParD(level)->isEvenTimestep,
																												para->getParD(level)->fluidNodeIndices,
        																										para->getParD(level)->numberOfFluidNodes);

	getLastCudaError("LB_Kernel_CumulantK17Almighty execution failed");
}

template<TurbulenceModel turbulenceModel>
void CumulantK17Almighty<turbulenceModel>::runOnIndices( const unsigned int *indices, unsigned int size_indices, CollisionTemplate collisionTemplate, int streamIndex )
{
	hipStream_t stream = (streamIndex == -1) ? hipStreamLegacy : para->getStreamManager()->getStream(streamIndex);

	switch (collisionTemplate)
	{
		case CollisionTemplate::Default:
			LB_Kernel_CumulantK17Almighty < turbulenceModel, false, false  > <<< cudaGrid.grid, cudaGrid.threads, 0, stream >>>(   	para->getParD(level)->omega, 	
																																	para->getParD(level)->typeOfGridNode, 										
																																	para->getParD(level)->neighborX, para->getParD(level)->neighborY, para->getParD(level)->neighborZ,	
																																	para->getParD(level)->distributions.f[0],	
																																	para->getParD(level)->rho,		
																																	para->getParD(level)->velocityX, para->getParD(level)->velocityY, para->getParD(level)->velocityZ,	
																																	para->getParD(level)->turbViscosity,
																																	para->getSGSConstant(),
																																	(unsigned long)para->getParD(level)->numberOfNodes,	
																																	level,				
																																	para->getIsBodyForce(),				
																																	para->getForcesDev(),				
																																	para->getParD(level)->forceX_SP, para->getParD(level)->forceY_SP, para->getParD(level)->forceZ_SP,
																																	para->getQuadricLimitersDev(),			
																																	para->getParD(level)->isEvenTimestep,
																																	indices,
																																	size_indices);
			break;
		
		case CollisionTemplate::WriteMacroVars:
			LB_Kernel_CumulantK17Almighty < turbulenceModel, true, false  > <<< cudaGrid.grid, cudaGrid.threads, 0, stream >>>( para->getParD(level)->omega, 	
																																para->getParD(level)->typeOfGridNode, 										
																																para->getParD(level)->neighborX, para->getParD(level)->neighborY, para->getParD(level)->neighborZ,	
																																para->getParD(level)->distributions.f[0],	
																																para->getParD(level)->rho,		
																																para->getParD(level)->velocityX, para->getParD(level)->velocityY, para->getParD(level)->velocityZ,	
																																para->getParD(level)->turbViscosity,
																																para->getSGSConstant(),
																																(unsigned long)para->getParD(level)->numberOfNodes,	
																																level,				
																																para->getIsBodyForce(),				
																																para->getForcesDev(),				
																																para->getParD(level)->forceX_SP, para->getParD(level)->forceY_SP, para->getParD(level)->forceZ_SP,
																																para->getQuadricLimitersDev(),			
																																para->getParD(level)->isEvenTimestep,
																																indices,
																																size_indices);
			break;
		
		case CollisionTemplate::AllFeatures:
			LB_Kernel_CumulantK17Almighty < turbulenceModel, true, true  > <<< cudaGrid.grid, cudaGrid.threads, 0, stream >>>(  para->getParD(level)->omega, 	
																																para->getParD(level)->typeOfGridNode, 										
																																para->getParD(level)->neighborX, para->getParD(level)->neighborY, para->getParD(level)->neighborZ,	
																																para->getParD(level)->distributions.f[0],	
																																para->getParD(level)->rho,		
																																para->getParD(level)->velocityX, para->getParD(level)->velocityY, para->getParD(level)->velocityZ,	
																																para->getParD(level)->turbViscosity,
																																para->getSGSConstant(),
																																(unsigned long)para->getParD(level)->numberOfNodes,	
																																level,				
																																para->getIsBodyForce(),				
																																para->getForcesDev(),				
																																para->getParD(level)->forceX_SP, para->getParD(level)->forceY_SP, para->getParD(level)->forceZ_SP,
																																para->getQuadricLimitersDev(),			
																																para->getParD(level)->isEvenTimestep,
																																indices,
																																size_indices);
			break;
		case CollisionTemplate::ApplyBodyForce:
			LB_Kernel_CumulantK17Almighty < turbulenceModel, false, true  > <<< cudaGrid.grid, cudaGrid.threads, 0, stream >>>( para->getParD(level)->omega, 	
																																para->getParD(level)->typeOfGridNode, 										
																																para->getParD(level)->neighborX, para->getParD(level)->neighborY, para->getParD(level)->neighborZ,	
																																para->getParD(level)->distributions.f[0],	
																																para->getParD(level)->rho,		
																																para->getParD(level)->velocityX, para->getParD(level)->velocityY, para->getParD(level)->velocityZ,	
																																para->getParD(level)->turbViscosity,
																																para->getSGSConstant(),
																																(unsigned long)para->getParD(level)->numberOfNodes,	
																																level,				
																																para->getIsBodyForce(),				
																																para->getForcesDev(),				
																																para->getParD(level)->forceX_SP, para->getParD(level)->forceY_SP, para->getParD(level)->forceZ_SP,
																																para->getQuadricLimitersDev(),			
																																para->getParD(level)->isEvenTimestep,
																																indices,
																																size_indices);
			break;
		default:
			throw std::runtime_error("Invalid CollisionTemplate in CumulantK17Almighty::runOnIndices()");
			break;
	}

	getLastCudaError("LB_Kernel_CumulantK17Almighty execution failed");
}

template<TurbulenceModel turbulenceModel>
CumulantK17Almighty<turbulenceModel>::CumulantK17Almighty(std::shared_ptr<Parameter> para, int level)
{
	this->para = para;
	this->level = level;

	myPreProcessorTypes.push_back(InitCompSP27);

	myKernelGroup = BasicKernel;

	this->cudaGrid = vf::cuda::CudaGrid(para->getParD(level)->numberofthreads, para->getParD(level)->numberOfNodes);
	this->kernelUsesFluidNodeIndices = true;
	
	VF_LOG_INFO("Using turbulence model: {}", turbulenceModel);
}

template class CumulantK17Almighty<TurbulenceModel::AMD>;
template class CumulantK17Almighty<TurbulenceModel::Smagorinsky>;
template class CumulantK17Almighty<TurbulenceModel::QR>;
template class CumulantK17Almighty<TurbulenceModel::None>;
