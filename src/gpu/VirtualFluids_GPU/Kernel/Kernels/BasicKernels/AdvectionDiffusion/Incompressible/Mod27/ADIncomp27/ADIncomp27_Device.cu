#include "hip/hip_runtime.h"
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;
#include "math.h"

__global__ void LB_Kernel_AD_Incomp_27(real diffusivity,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* DDStart,
	real* DD27,
	int size_Mat,
	bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if ((BC != GEO_SOLID) && (BC != GEO_VOID))
		{
			Distributions27 D;
			if (EvenOrOdd == true)
			{
				D.f[DIR_P00] = &DDStart[DIR_P00   *size_Mat];
				D.f[DIR_M00] = &DDStart[DIR_M00   *size_Mat];
				D.f[DIR_0P0] = &DDStart[DIR_0P0   *size_Mat];
				D.f[DIR_0M0] = &DDStart[DIR_0M0   *size_Mat];
				D.f[DIR_00P] = &DDStart[DIR_00P   *size_Mat];
				D.f[DIR_00M] = &DDStart[DIR_00M   *size_Mat];
				D.f[DIR_PP0] = &DDStart[DIR_PP0  *size_Mat];
				D.f[DIR_MM0] = &DDStart[DIR_MM0  *size_Mat];
				D.f[DIR_PM0] = &DDStart[DIR_PM0  *size_Mat];
				D.f[DIR_MP0] = &DDStart[DIR_MP0  *size_Mat];
				D.f[DIR_P0P] = &DDStart[DIR_P0P  *size_Mat];
				D.f[DIR_M0M] = &DDStart[DIR_M0M  *size_Mat];
				D.f[DIR_P0M] = &DDStart[DIR_P0M  *size_Mat];
				D.f[DIR_M0P] = &DDStart[DIR_M0P  *size_Mat];
				D.f[DIR_0PP] = &DDStart[DIR_0PP  *size_Mat];
				D.f[DIR_0MM] = &DDStart[DIR_0MM  *size_Mat];
				D.f[DIR_0PM] = &DDStart[DIR_0PM  *size_Mat];
				D.f[DIR_0MP] = &DDStart[DIR_0MP  *size_Mat];
				D.f[DIR_000] = &DDStart[DIR_000*size_Mat];
				D.f[DIR_PPP] = &DDStart[DIR_PPP *size_Mat];
				D.f[DIR_MMP] = &DDStart[DIR_MMP *size_Mat];
				D.f[DIR_PMP] = &DDStart[DIR_PMP *size_Mat];
				D.f[DIR_MPP] = &DDStart[DIR_MPP *size_Mat];
				D.f[DIR_PPM] = &DDStart[DIR_PPM *size_Mat];
				D.f[DIR_MMM] = &DDStart[DIR_MMM *size_Mat];
				D.f[DIR_PMM] = &DDStart[DIR_PMM *size_Mat];
				D.f[DIR_MPM] = &DDStart[DIR_MPM *size_Mat];
			}
			else
			{
				D.f[DIR_M00] = &DDStart[DIR_P00   *size_Mat];
				D.f[DIR_P00] = &DDStart[DIR_M00   *size_Mat];
				D.f[DIR_0M0] = &DDStart[DIR_0P0   *size_Mat];
				D.f[DIR_0P0] = &DDStart[DIR_0M0   *size_Mat];
				D.f[DIR_00M] = &DDStart[DIR_00P   *size_Mat];
				D.f[DIR_00P] = &DDStart[DIR_00M   *size_Mat];
				D.f[DIR_MM0] = &DDStart[DIR_PP0  *size_Mat];
				D.f[DIR_PP0] = &DDStart[DIR_MM0  *size_Mat];
				D.f[DIR_MP0] = &DDStart[DIR_PM0  *size_Mat];
				D.f[DIR_PM0] = &DDStart[DIR_MP0  *size_Mat];
				D.f[DIR_M0M] = &DDStart[DIR_P0P  *size_Mat];
				D.f[DIR_P0P] = &DDStart[DIR_M0M  *size_Mat];
				D.f[DIR_M0P] = &DDStart[DIR_P0M  *size_Mat];
				D.f[DIR_P0M] = &DDStart[DIR_M0P  *size_Mat];
				D.f[DIR_0MM] = &DDStart[DIR_0PP  *size_Mat];
				D.f[DIR_0PP] = &DDStart[DIR_0MM  *size_Mat];
				D.f[DIR_0MP] = &DDStart[DIR_0PM  *size_Mat];
				D.f[DIR_0PM] = &DDStart[DIR_0MP  *size_Mat];
				D.f[DIR_000] = &DDStart[DIR_000*size_Mat];
				D.f[DIR_MMM] = &DDStart[DIR_PPP *size_Mat];
				D.f[DIR_PPM] = &DDStart[DIR_MMP *size_Mat];
				D.f[DIR_MPM] = &DDStart[DIR_PMP *size_Mat];
				D.f[DIR_PMM] = &DDStart[DIR_MPP *size_Mat];
				D.f[DIR_MMP] = &DDStart[DIR_PPM *size_Mat];
				D.f[DIR_PPP] = &DDStart[DIR_MMM *size_Mat];
				D.f[DIR_MPP] = &DDStart[DIR_PMM *size_Mat];
				D.f[DIR_PMP] = &DDStart[DIR_MPM *size_Mat];
			}

			Distributions27 D27;
			if (EvenOrOdd == true)
			{
				D27.f[DIR_P00] = &DD27[DIR_P00   *size_Mat];
				D27.f[DIR_M00] = &DD27[DIR_M00   *size_Mat];
				D27.f[DIR_0P0] = &DD27[DIR_0P0   *size_Mat];
				D27.f[DIR_0M0] = &DD27[DIR_0M0   *size_Mat];
				D27.f[DIR_00P] = &DD27[DIR_00P   *size_Mat];
				D27.f[DIR_00M] = &DD27[DIR_00M   *size_Mat];
				D27.f[DIR_PP0] = &DD27[DIR_PP0  *size_Mat];
				D27.f[DIR_MM0] = &DD27[DIR_MM0  *size_Mat];
				D27.f[DIR_PM0] = &DD27[DIR_PM0  *size_Mat];
				D27.f[DIR_MP0] = &DD27[DIR_MP0  *size_Mat];
				D27.f[DIR_P0P] = &DD27[DIR_P0P  *size_Mat];
				D27.f[DIR_M0M] = &DD27[DIR_M0M  *size_Mat];
				D27.f[DIR_P0M] = &DD27[DIR_P0M  *size_Mat];
				D27.f[DIR_M0P] = &DD27[DIR_M0P  *size_Mat];
				D27.f[DIR_0PP] = &DD27[DIR_0PP  *size_Mat];
				D27.f[DIR_0MM] = &DD27[DIR_0MM  *size_Mat];
				D27.f[DIR_0PM] = &DD27[DIR_0PM  *size_Mat];
				D27.f[DIR_0MP] = &DD27[DIR_0MP  *size_Mat];
				D27.f[DIR_000] = &DD27[DIR_000*size_Mat];
				D27.f[DIR_PPP] = &DD27[DIR_PPP *size_Mat];
				D27.f[DIR_MMP] = &DD27[DIR_MMP *size_Mat];
				D27.f[DIR_PMP] = &DD27[DIR_PMP *size_Mat];
				D27.f[DIR_MPP] = &DD27[DIR_MPP *size_Mat];
				D27.f[DIR_PPM] = &DD27[DIR_PPM *size_Mat];
				D27.f[DIR_MMM] = &DD27[DIR_MMM *size_Mat];
				D27.f[DIR_PMM] = &DD27[DIR_PMM *size_Mat];
				D27.f[DIR_MPM] = &DD27[DIR_MPM *size_Mat];
			}
			else
			{
				D27.f[DIR_M00] = &DD27[DIR_P00   *size_Mat];
				D27.f[DIR_P00] = &DD27[DIR_M00   *size_Mat];
				D27.f[DIR_0M0] = &DD27[DIR_0P0   *size_Mat];
				D27.f[DIR_0P0] = &DD27[DIR_0M0   *size_Mat];
				D27.f[DIR_00M] = &DD27[DIR_00P   *size_Mat];
				D27.f[DIR_00P] = &DD27[DIR_00M   *size_Mat];
				D27.f[DIR_MM0] = &DD27[DIR_PP0  *size_Mat];
				D27.f[DIR_PP0] = &DD27[DIR_MM0  *size_Mat];
				D27.f[DIR_MP0] = &DD27[DIR_PM0  *size_Mat];
				D27.f[DIR_PM0] = &DD27[DIR_MP0  *size_Mat];
				D27.f[DIR_M0M] = &DD27[DIR_P0P  *size_Mat];
				D27.f[DIR_P0P] = &DD27[DIR_M0M  *size_Mat];
				D27.f[DIR_M0P] = &DD27[DIR_P0M  *size_Mat];
				D27.f[DIR_P0M] = &DD27[DIR_M0P  *size_Mat];
				D27.f[DIR_0MM] = &DD27[DIR_0PP  *size_Mat];
				D27.f[DIR_0PP] = &DD27[DIR_0MM  *size_Mat];
				D27.f[DIR_0MP] = &DD27[DIR_0PM  *size_Mat];
				D27.f[DIR_0PM] = &DD27[DIR_0MP  *size_Mat];
				D27.f[DIR_000] = &DD27[DIR_000*size_Mat];
				D27.f[DIR_MMM] = &DD27[DIR_PPP *size_Mat];
				D27.f[DIR_PPM] = &DD27[DIR_MMP *size_Mat];
				D27.f[DIR_MPM] = &DD27[DIR_PMP *size_Mat];
				D27.f[DIR_PMM] = &DD27[DIR_MPP *size_Mat];
				D27.f[DIR_MMP] = &DD27[DIR_PPM *size_Mat];
				D27.f[DIR_PPP] = &DD27[DIR_MMM *size_Mat];
				D27.f[DIR_MPP] = &DD27[DIR_PMM *size_Mat];
				D27.f[DIR_PMP] = &DD27[DIR_MPM *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			unsigned int kw = neighborX[k];
			unsigned int ks = neighborY[k];
			unsigned int kb = neighborZ[k];
			unsigned int ksw = neighborY[kw];
			unsigned int kbw = neighborZ[kw];
			unsigned int kbs = neighborZ[ks];
			unsigned int kbsw = neighborZ[ksw];
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real fW = (D.f[DIR_P00])[k];//ke
			real fE = (D.f[DIR_M00])[kw];
			real fS = (D.f[DIR_0P0])[k];//kn
			real fN = (D.f[DIR_0M0])[ks];
			real fB = (D.f[DIR_00P])[k];//kt
			real fT = (D.f[DIR_00M])[kb];
			real fSW = (D.f[DIR_PP0])[k];//kne
			real fNE = (D.f[DIR_MM0])[ksw];
			real fNW = (D.f[DIR_PM0])[ks];//kse
			real fSE = (D.f[DIR_MP0])[kw];//knw
			real fBW = (D.f[DIR_P0P])[k];//kte
			real fTE = (D.f[DIR_M0M])[kbw];
			real fTW = (D.f[DIR_P0M])[kb];//kbe
			real fBE = (D.f[DIR_M0P])[kw];//ktw
			real fBS = (D.f[DIR_0PP])[k];//ktn
			real fTN = (D.f[DIR_0MM])[kbs];
			real fTS = (D.f[DIR_0PM])[kb];//kbn
			real fBN = (D.f[DIR_0MP])[ks];//kts
			//real fZERO = (D.f[DIR_000])[k];//kzero
			real fBSW = (D.f[DIR_PPP])[k];//ktne
			real fBNE = (D.f[DIR_MMP])[ksw];//ktsw
			real fBNW = (D.f[DIR_PMP])[ks];//ktse
			real fBSE = (D.f[DIR_MPP])[kw];//ktnw
			real fTSW = (D.f[DIR_PPM])[kb];//kbne
			real fTNE = (D.f[DIR_MMM])[kbsw];
			real fTNW = (D.f[DIR_PMM])[kbs];//kbse
			real fTSE = (D.f[DIR_MPM])[kbw];//kbnw
										   ////////////////////////////////////////////////////////////////////////////////
										   //real f27E    =  (D27.f[DIR_P00   ])[k  ];//ke
										   //real f27W    =  (D27.f[DIR_M00   ])[kw ];
										   //real f27N    =  (D27.f[DIR_0P0   ])[k  ];//kn
										   //real f27S    =  (D27.f[DIR_0M0   ])[ks ];
										   //real f27T    =  (D27.f[DIR_00P   ])[k  ];//kt
										   //real f27B    =  (D27.f[DIR_00M   ])[kb ];
										   //real f27NE   =  (D27.f[DIR_PP0  ])[k  ];//kne
										   //real f27SW   =  (D27.f[DIR_MM0  ])[ksw];
										   //real f27SE   =  (D27.f[DIR_PM0  ])[ks ];//kse
										   //real f27NW   =  (D27.f[DIR_MP0  ])[kw ];//knw
										   //real f27TE   =  (D27.f[DIR_P0P  ])[k  ];//kte
										   //real f27BW   =  (D27.f[DIR_M0M  ])[kbw];
										   //real f27BE   =  (D27.f[DIR_P0M  ])[kb ];//kbe
										   //real f27TW   =  (D27.f[DIR_M0P  ])[kw ];//ktw
										   //real f27TN   =  (D27.f[DIR_0PP  ])[k  ];//ktn
										   //real f27BS   =  (D27.f[DIR_0MM  ])[kbs];
										   //real f27BN   =  (D27.f[DIR_0PM  ])[kb ];//kbn
										   //real f27TS   =  (D27.f[DIR_0MP  ])[ks ];//kts
										   //real f27ZERO =  (D27.f[DIR_000])[k  ];//kzero
										   //real f27TNE  =  (D27.f[DIR_PPP ])[k  ];//ktne
										   //real f27TSW  =  (D27.f[DIR_MMP ])[ksw];//ktsw
										   //real f27TSE  =  (D27.f[DIR_PMP ])[ks ];//ktse
										   //real f27TNW  =  (D27.f[DIR_MPP ])[kw ];//ktnw
										   //real f27BNE  =  (D27.f[DIR_PPM ])[kb ];//kbne
										   //real f27BSW  =  (D27.f[DIR_MMM ])[kbsw];
										   //real f27BSE  =  (D27.f[DIR_PMM ])[kbs];//kbse
										   //real f27BNW  =  (D27.f[DIR_MPM ])[kbw];//kbnw
										   ////////////////////////////////////////////////////////////////////////////////
										   //real vx1     =  ((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
										   //real vx2     =  ((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
										   //real vx3     =  ((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
										   ////////////////////////////////////////////////////////////////////////////////


			real mfcbb = (D27.f[DIR_P00])[k];
			real mfabb = (D27.f[DIR_M00])[kw];
			real mfbcb = (D27.f[DIR_0P0])[k];
			real mfbab = (D27.f[DIR_0M0])[ks];
			real mfbbc = (D27.f[DIR_00P])[k];
			real mfbba = (D27.f[DIR_00M])[kb];
			real mfccb = (D27.f[DIR_PP0])[k];
			real mfaab = (D27.f[DIR_MM0])[ksw];
			real mfcab = (D27.f[DIR_PM0])[ks];
			real mfacb = (D27.f[DIR_MP0])[kw];
			real mfcbc = (D27.f[DIR_P0P])[k];
			real mfaba = (D27.f[DIR_M0M])[kbw];
			real mfcba = (D27.f[DIR_P0M])[kb];
			real mfabc = (D27.f[DIR_M0P])[kw];
			real mfbcc = (D27.f[DIR_0PP])[k];
			real mfbaa = (D27.f[DIR_0MM])[kbs];
			real mfbca = (D27.f[DIR_0PM])[kb];
			real mfbac = (D27.f[DIR_0MP])[ks];
			real mfbbb = (D27.f[DIR_000])[k];
			real mfccc = (D27.f[DIR_PPP])[k];
			real mfaac = (D27.f[DIR_MMP])[ksw];
			real mfcac = (D27.f[DIR_PMP])[ks];
			real mfacc = (D27.f[DIR_MPP])[kw];
			real mfcca = (D27.f[DIR_PPM])[kb];
			real mfaaa = (D27.f[DIR_MMM])[kbsw];
			real mfcaa = (D27.f[DIR_PMM])[kbs];
			real mfaca = (D27.f[DIR_MPM])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			//Conc
			real drho = ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				((mfabb + mfcbb) + (mfbab + mfbcb)) + (mfbba + mfbbc)) + mfbbb;
			//real rho = c1o1 + drho;
			////////////////////////////////////////////////////////////////////////////////////

			real vvx = ((fTNE - fBSW) + (fBNE - fTSW) + (fTSE - fBNW) + (fBSE - fTNW) + (fNE - fSW) + (fSE - fNW) + (fTE - fBW) + (fBE - fTW) + (fE - fW));
			real vvy = ((fTNE - fBSW) + (fBNE - fTSW) + (fBNW - fTSE) + (fTNW - fBSE) + (fNE - fSW) + (fNW - fSE) + (fTN - fBS) + (fBN - fTS) + (fN - fS));
			real vvz = ((fTNE - fBSW) + (fTSW - fBNE) + (fTSE - fBNW) + (fTNW - fBSE) + (fTE - fBW) + (fTW - fBE) + (fTN - fBS) + (fTS - fBN) + (fT - fB));
			////////////////////////////////////////////////////////////////////////////////
			real omegaD = c2o1 / (c6o1 * diffusivity + c1o1);
			////real omegaD     = -three + sqrt(three);
			////real Lam         = -(c1o2+one/omegaD);
			////real nue_d       = Lam/three;
			//real ae          = zero;
			////real ae          = diffusivity/nue_d - one;
			//real ux_sq       = vx * vx;
			//real uy_sq       = vy * vy;
			//real uz_sq       = vz * vz;


			////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			//D3Q7
			//real ConcD       = f7ZERO+f7E+f7W+f7N+f7S+f7T+f7B;
			//(D7.f[0])[k  ] = f7ZERO*(one+omegaD)-omegaD*ConcD*(c1o3*(ae*(-three))-(ux_sq+uy_sq+uz_sq));
			//(D7.f[2])[kw ] = f7E   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)+vx*c1o2);
			//(D7.f[1])[k  ] = f7W   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)-vx*c1o2);
			//(D7.f[4])[ks ] = f7N   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)+vy*c1o2);
			//(D7.f[3])[k  ] = f7S   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)-vy*c1o2);
			//(D7.f[6])[kb ] = f7T   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)+vz*c1o2);
			//(D7.f[5])[k  ] = f7B   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)-vz*c1o2);
			////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			////D3Q27
			//real ConcD   = (f27TNE+f27BSW)+(f27TSW+f27BNE)+(f27TSE+f27BNW)+(f27TNW+f27BSE)+
			//                  (f27NE+f27SW)+(f27NW+f27SE)+(f27TE+f27BW)+(f27BE+f27TW)+(f27TN+f27BS)+(f27BN+f27TS)+
			//                  (f27E+f27W)+(f27N+f27S)+(f27T+f27B)+f27ZERO;
			//real cusq    =  c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

			//(D27.f[ DIR_P00   ])[k   ] = f27W    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cusq);                                                                     
			//(D27.f[ DIR_M00   ])[kw  ] = f27E    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cusq);                                                                     
			//(D27.f[ DIR_0P0   ])[k   ] = f27S    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cusq);
			//(D27.f[ DIR_0M0   ])[ks  ] = f27N    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cusq);
			//(D27.f[ DIR_00P   ])[k   ] = f27B    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cusq);
			//(D27.f[ DIR_00M   ])[kb  ] = f27T    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cusq);
			//(D27.f[ DIR_PP0  ])[k   ] = f27SW   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cusq);
			//(D27.f[ DIR_MM0  ])[ksw ] = f27NE   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cusq);
			//(D27.f[ DIR_PM0  ])[ks  ] = f27NW   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cusq);
			//(D27.f[ DIR_MP0  ])[kw  ] = f27SE   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cusq);
			//(D27.f[ DIR_P0P  ])[k   ] = f27BW   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cusq);
			//(D27.f[ DIR_M0M  ])[kbw ] = f27TE   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cusq);
			//(D27.f[ DIR_P0M  ])[kb  ] = f27TW   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cusq);
			//(D27.f[ DIR_M0P  ])[kw  ] = f27BE   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cusq);
			//(D27.f[ DIR_0PP  ])[k   ] = f27BS   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cusq);
			//(D27.f[ DIR_0MM  ])[kbs ] = f27TN   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cusq);
			//(D27.f[ DIR_0PM  ])[kb  ] = f27TS   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cusq);
			//(D27.f[ DIR_0MP  ])[ks  ] = f27BN   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cusq);
			//(D27.f[ DIR_000])[k   ] = f27ZERO *(one-omegaD)+omegaD* c8over27* ConcD*(one-cusq);
			//(D27.f[ DIR_PPP ])[k   ] = f27BSW  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cusq);
			//(D27.f[ DIR_PMP ])[ks  ] = f27BNW  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cusq);
			//(D27.f[ DIR_PPM ])[kb  ] = f27TSW  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cusq);
			//(D27.f[ DIR_PMM ])[kbs ] = f27TNW  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cusq);
			//(D27.f[ DIR_MPP ])[kw  ] = f27BSE  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cusq);
			//(D27.f[ DIR_MMP ])[ksw ] = f27BNE  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cusq);
			//(D27.f[ DIR_MPM ])[kbw ] = f27TSE  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cusq);
			//(D27.f[ DIR_MMM ])[kbsw] = f27TNE  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cusq);
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			real oMdrho = c0o1;//one; // comp special
			real m0, m1, m2;
			real vx2 = vvx*vvx;
			real vy2 = vvy*vvy;
			real vz2 = vvz*vvz;

			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m2 = mfaaa + mfaac;
			m1 = mfaac - mfaaa;
			m0 = m2 + mfaab;
			mfaaa = m0;
			m0 += c1o36 * oMdrho;
			mfaab = m1 - m0 * vvz;
			mfaac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaba + mfabc;
			m1 = mfabc - mfaba;
			m0 = m2 + mfabb;
			mfaba = m0;
			m0 += c1o9 * oMdrho;
			mfabb = m1 - m0 * vvz;
			mfabc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaca + mfacc;
			m1 = mfacc - mfaca;
			m0 = m2 + mfacb;
			mfaca = m0;
			m0 += c1o36 * oMdrho;
			mfacb = m1 - m0 * vvz;
			mfacc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbaa + mfbac;
			m1 = mfbac - mfbaa;
			m0 = m2 + mfbab;
			mfbaa = m0;
			m0 += c1o9 * oMdrho;
			mfbab = m1 - m0 * vvz;
			mfbac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbba + mfbbc;
			m1 = mfbbc - mfbba;
			m0 = m2 + mfbbb;
			mfbba = m0;
			m0 += c4o9 * oMdrho;
			mfbbb = m1 - m0 * vvz;
			mfbbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbca + mfbcc;
			m1 = mfbcc - mfbca;
			m0 = m2 + mfbcb;
			mfbca = m0;
			m0 += c1o9 * oMdrho;
			mfbcb = m1 - m0 * vvz;
			mfbcc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcaa + mfcac;
			m1 = mfcac - mfcaa;
			m0 = m2 + mfcab;
			mfcaa = m0;
			m0 += c1o36 * oMdrho;
			mfcab = m1 - m0 * vvz;
			mfcac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcba + mfcbc;
			m1 = mfcbc - mfcba;
			m0 = m2 + mfcbb;
			mfcba = m0;
			m0 += c1o9 * oMdrho;
			mfcbb = m1 - m0 * vvz;
			mfcbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcca + mfccc;
			m1 = mfccc - mfcca;
			m0 = m2 + mfccb;
			mfcca = m0;
			m0 += c1o36 * oMdrho;
			mfccb = m1 - m0 * vvz;
			mfccc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2 = mfaaa + mfaca;
			m1 = mfaca - mfaaa;
			m0 = m2 + mfaba;
			mfaaa = m0;
			m0 += c1o6 * oMdrho;
			mfaba = m1 - m0 * vvy;
			mfaca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaab + mfacb;
			m1 = mfacb - mfaab;
			m0 = m2 + mfabb;
			mfaab = m0;
			mfabb = m1 - m0 * vvy;
			mfacb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaac + mfacc;
			m1 = mfacc - mfaac;
			m0 = m2 + mfabc;
			mfaac = m0;
			m0 += c1o18 * oMdrho;
			mfabc = m1 - m0 * vvy;
			mfacc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbaa + mfbca;
			m1 = mfbca - mfbaa;
			m0 = m2 + mfbba;
			mfbaa = m0;
			m0 += c2o3 * oMdrho;
			mfbba = m1 - m0 * vvy;
			mfbca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbab + mfbcb;
			m1 = mfbcb - mfbab;
			m0 = m2 + mfbbb;
			mfbab = m0;
			mfbbb = m1 - m0 * vvy;
			mfbcb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbac + mfbcc;
			m1 = mfbcc - mfbac;
			m0 = m2 + mfbbc;
			mfbac = m0;
			m0 += c2o9 * oMdrho;
			mfbbc = m1 - m0 * vvy;
			mfbcc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcaa + mfcca;
			m1 = mfcca - mfcaa;
			m0 = m2 + mfcba;
			mfcaa = m0;
			m0 += c1o6 * oMdrho;
			mfcba = m1 - m0 * vvy;
			mfcca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcab + mfccb;
			m1 = mfccb - mfcab;
			m0 = m2 + mfcbb;
			mfcab = m0;
			mfcbb = m1 - m0 * vvy;
			mfccb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcac + mfccc;
			m1 = mfccc - mfcac;
			m0 = m2 + mfcbc;
			mfcac = m0;
			m0 += c1o18 * oMdrho;
			mfcbc = m1 - m0 * vvy;
			mfccc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2 = mfaaa + mfcaa;
			m1 = mfcaa - mfaaa;
			m0 = m2 + mfbaa;
			mfaaa = m0;
			m0 += c1o1* oMdrho;
			mfbaa = m1 - m0 * vvx;
			mfcaa = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaba + mfcba;
			m1 = mfcba - mfaba;
			m0 = m2 + mfbba;
			mfaba = m0;
			mfbba = m1 - m0 * vvx;
			mfcba = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaca + mfcca;
			m1 = mfcca - mfaca;
			m0 = m2 + mfbca;
			mfaca = m0;
			m0 += c1o3 * oMdrho;
			mfbca = m1 - m0 * vvx;
			mfcca = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaab + mfcab;
			m1 = mfcab - mfaab;
			m0 = m2 + mfbab;
			mfaab = m0;
			mfbab = m1 - m0 * vvx;
			mfcab = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfabb + mfcbb;
			m1 = mfcbb - mfabb;
			m0 = m2 + mfbbb;
			mfabb = m0;
			mfbbb = m1 - m0 * vvx;
			mfcbb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfacb + mfccb;
			m1 = mfccb - mfacb;
			m0 = m2 + mfbcb;
			mfacb = m0;
			mfbcb = m1 - m0 * vvx;
			mfccb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaac + mfcac;
			m1 = mfcac - mfaac;
			m0 = m2 + mfbac;
			mfaac = m0;
			m0 += c1o3 * oMdrho;
			mfbac = m1 - m0 * vvx;
			mfcac = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfabc + mfcbc;
			m1 = mfcbc - mfabc;
			m0 = m2 + mfbbc;
			mfabc = m0;
			mfbbc = m1 - m0 * vvx;
			mfcbc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfacc + mfccc;
			m1 = mfccc - mfacc;
			m0 = m2 + mfbcc;
			mfacc = m0;
			m0 += c1o9 * oMdrho;
			mfbcc = m1 - m0 * vvx;
			mfccc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////

			//if(mfaaa < zero) omegaD = one;
			real limit = c9o1*omegaD*omegaD*(mfbaa*mfbaa + mfaba*mfaba + mfaab*mfaab);
			//real CC=c1o2;
			//if ((two*mfaaa*mfaaa<limit)) omegaD=two / (six * (diffusivity+((limit/(1.0e-10f+two*mfaaa*mfaaa)-one)*(c1o6-diffusivity))*c1o2) + one);
			if ((c2o1*mfaaa*mfaaa<limit)) omegaD = c1o1;
			//omegaD = two / (six * (diffusivity+CC*limit) + one);

			//mfaaa = c1o2;
			//trans 3.
			real Mabc = mfabc - mfaba*c1o3;
			real Mbca = mfbca - mfbaa*c1o3;
			real Macb = mfacb - mfaab*c1o3;
			real Mcba = mfcba - mfaba*c1o3;
			real Mcab = mfcab - mfaab*c1o3;
			real Mbac = mfbac - mfbaa*c1o3;
			//trans 5.
			real Mcbc = mfcbc - mfaba*c1o9;
			real Mbcc = mfbcc - mfbaa*c1o9;
			real Mccb = mfccb - mfaab*c1o9;

			//1.
			mfbaa *= c1o1 - omegaD;
			mfaba *= c1o1 - omegaD;
			mfaab *= c1o1 - omegaD;

			//3.
			//mfbca *= one - omegaD;
			//mfbac *= one - omegaD;
			//mfcba *= one - omegaD;
			//mfabc *= one - omegaD;
			//mfcab *= one - omegaD;
			//mfacb *= one - omegaD;

			//mfbbb *= one - omegaD; 
			Mabc = c0o1;
			Mbca = c0o1;
			Macb = c0o1;
			Mcba = c0o1;
			Mcab = c0o1;
			Mbac = c0o1;
			mfbbb = c0o1;

			//5.
			//mfbcc *= one - omegaD;
			//mfcbc *= one - omegaD;
			//mfccb *= one - omegaD;
			Mcbc = c0o1;
			Mbcc = c0o1;
			Mccb = c0o1;

			//2.
			mfbba = c0o1;
			mfbab = c0o1;
			mfabb = c0o1;

			mfcaa = c1o3 * drho;
			mfaca = c1o3 * drho;
			mfaac = c1o3 * drho;

			//4.
			mfacc = c1o9 * drho;
			mfcac = c1o9 * drho;
			mfcca = c1o9 * drho;

			mfcbb = c0o1;
			mfbcb = c0o1;
			mfbbc = c0o1;

			//6.
			mfccc = c1o27 * drho;

			//3.
			mfabc = Mabc + mfaba*c1o3;
			mfbca = Mbca + mfbaa*c1o3;
			mfacb = Macb + mfaab*c1o3;
			mfcba = Mcba + mfaba*c1o3;
			mfcab = Mcab + mfaab*c1o3;
			mfbac = Mbac + mfbaa*c1o3;
			//5.	  
			mfcbc = Mcbc + mfaba*c1o9;
			mfbcc = Mbcc + mfbaa*c1o9;
			mfccb = Mccb + mfaab*c1o9;

			////////////////////////////////////////////////////////////////////////////////////
			//back
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m0 = mfaac * c1o2 + mfaab * (vvz - c1o2) + (mfaaa + c1o1* oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfaac - c2o1* mfaab *  vvz + mfaaa                * (c1o1 - vz2) - c1o1* oMdrho * vz2;
			m2 = mfaac * c1o2 + mfaab * (vvz + c1o2) + (mfaaa + c1o1* oMdrho) * (vz2 + vvz) * c1o2;
			mfaaa = m0;
			mfaab = m1;
			mfaac = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfabc * c1o2 + mfabb * (vvz - c1o2) + mfaba * (vz2 - vvz) * c1o2;
			m1 = -mfabc - c2o1* mfabb *  vvz + mfaba * (c1o1 - vz2);
			m2 = mfabc * c1o2 + mfabb * (vvz + c1o2) + mfaba * (vz2 + vvz) * c1o2;
			mfaba = m0;
			mfabb = m1;
			mfabc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacc * c1o2 + mfacb * (vvz - c1o2) + (mfaca + c1o3 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfacc - c2o1* mfacb *  vvz + mfaca                  * (c1o1 - vz2) - c1o3 * oMdrho * vz2;
			m2 = mfacc * c1o2 + mfacb * (vvz + c1o2) + (mfaca + c1o3 * oMdrho) * (vz2 + vvz) * c1o2;
			mfaca = m0;
			mfacb = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfbac * c1o2 + mfbab * (vvz - c1o2) + mfbaa * (vz2 - vvz) * c1o2;
			m1 = -mfbac - c2o1* mfbab *  vvz + mfbaa * (c1o1 - vz2);
			m2 = mfbac * c1o2 + mfbab * (vvz + c1o2) + mfbaa * (vz2 + vvz) * c1o2;
			mfbaa = m0;
			mfbab = m1;
			mfbac = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbbc * c1o2 + mfbbb * (vvz - c1o2) + mfbba * (vz2 - vvz) * c1o2;
			m1 = -mfbbc - c2o1* mfbbb *  vvz + mfbba * (c1o1 - vz2);
			m2 = mfbbc * c1o2 + mfbbb * (vvz + c1o2) + mfbba * (vz2 + vvz) * c1o2;
			mfbba = m0;
			mfbbb = m1;
			mfbbc = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcc * c1o2 + mfbcb * (vvz - c1o2) + mfbca * (vz2 - vvz) * c1o2;
			m1 = -mfbcc - c2o1* mfbcb *  vvz + mfbca * (c1o1 - vz2);
			m2 = mfbcc * c1o2 + mfbcb * (vvz + c1o2) + mfbca * (vz2 + vvz) * c1o2;
			mfbca = m0;
			mfbcb = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcac * c1o2 + mfcab * (vvz - c1o2) + (mfcaa + c1o3 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfcac - c2o1* mfcab *  vvz + mfcaa                  * (c1o1 - vz2) - c1o3 * oMdrho * vz2;
			m2 = mfcac * c1o2 + mfcab * (vvz + c1o2) + (mfcaa + c1o3 * oMdrho) * (vz2 + vvz) * c1o2;
			mfcaa = m0;
			mfcab = m1;
			mfcac = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfcbc * c1o2 + mfcbb * (vvz - c1o2) + mfcba * (vz2 - vvz) * c1o2;
			m1 = -mfcbc - c2o1* mfcbb *  vvz + mfcba * (c1o1 - vz2);
			m2 = mfcbc * c1o2 + mfcbb * (vvz + c1o2) + mfcba * (vz2 + vvz) * c1o2;
			mfcba = m0;
			mfcbb = m1;
			mfcbc = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfccb * (vvz - c1o2) + (mfcca + c1o9 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfccc - c2o1* mfccb *  vvz + mfcca                  * (c1o1 - vz2) - c1o9 * oMdrho * vz2;
			m2 = mfccc * c1o2 + mfccb * (vvz + c1o2) + (mfcca + c1o9 * oMdrho) * (vz2 + vvz) * c1o2;
			mfcca = m0;
			mfccb = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m0 = mfaca * c1o2 + mfaba * (vvy - c1o2) + (mfaaa + c1o6 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfaca - c2o1* mfaba *  vvy + mfaaa                  * (c1o1 - vy2) - c1o6 * oMdrho * vy2;
			m2 = mfaca * c1o2 + mfaba * (vvy + c1o2) + (mfaaa + c1o6 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaaa = m0;
			mfaba = m1;
			mfaca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacb * c1o2 + mfabb * (vvy - c1o2) + (mfaab + c2o3 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfacb - c2o1* mfabb *  vvy + mfaab                  * (c1o1 - vy2) - c2o3 * oMdrho * vy2;
			m2 = mfacb * c1o2 + mfabb * (vvy + c1o2) + (mfaab + c2o3 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaab = m0;
			mfabb = m1;
			mfacb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacc * c1o2 + mfabc * (vvy - c1o2) + (mfaac + c1o6 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfacc - c2o1* mfabc *  vvy + mfaac                  * (c1o1 - vy2) - c1o6 * oMdrho * vy2;
			m2 = mfacc * c1o2 + mfabc * (vvy + c1o2) + (mfaac + c1o6 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaac = m0;
			mfabc = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfbca * c1o2 + mfbba * (vvy - c1o2) + mfbaa * (vy2 - vvy) * c1o2;
			m1 = -mfbca - c2o1* mfbba *  vvy + mfbaa * (c1o1 - vy2);
			m2 = mfbca * c1o2 + mfbba * (vvy + c1o2) + mfbaa * (vy2 + vvy) * c1o2;
			mfbaa = m0;
			mfbba = m1;
			mfbca = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcb * c1o2 + mfbbb * (vvy - c1o2) + mfbab * (vy2 - vvy) * c1o2;
			m1 = -mfbcb - c2o1* mfbbb *  vvy + mfbab * (c1o1 - vy2);
			m2 = mfbcb * c1o2 + mfbbb * (vvy + c1o2) + mfbab * (vy2 + vvy) * c1o2;
			mfbab = m0;
			mfbbb = m1;
			mfbcb = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcc * c1o2 + mfbbc * (vvy - c1o2) + mfbac * (vy2 - vvy) * c1o2;
			m1 = -mfbcc - c2o1* mfbbc *  vvy + mfbac * (c1o1 - vy2);
			m2 = mfbcc * c1o2 + mfbbc * (vvy + c1o2) + mfbac * (vy2 + vvy) * c1o2;
			mfbac = m0;
			mfbbc = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcca * c1o2 + mfcba * (vvy - c1o2) + (mfcaa + c1o18 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfcca - c2o1* mfcba *  vvy + mfcaa                   * (c1o1 - vy2) - c1o18 * oMdrho * vy2;
			m2 = mfcca * c1o2 + mfcba * (vvy + c1o2) + (mfcaa + c1o18 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcaa = m0;
			mfcba = m1;
			mfcca = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccb * c1o2 + mfcbb * (vvy - c1o2) + (mfcab + c2o9 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfccb - c2o1* mfcbb *  vvy + mfcab                  * (c1o1 - vy2) - c2o9 * oMdrho * vy2;
			m2 = mfccb * c1o2 + mfcbb * (vvy + c1o2) + (mfcab + c2o9 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcab = m0;
			mfcbb = m1;
			mfccb = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfcbc * (vvy - c1o2) + (mfcac + c1o18 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfccc - c2o1* mfcbc *  vvy + mfcac                   * (c1o1 - vy2) - c1o18 * oMdrho * vy2;
			m2 = mfccc * c1o2 + mfcbc * (vvy + c1o2) + (mfcac + c1o18 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcac = m0;
			mfcbc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m0 = mfcaa * c1o2 + mfbaa * (vvx - c1o2) + (mfaaa + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcaa - c2o1* mfbaa *  vvx + mfaaa                   * (c1o1 - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcaa * c1o2 + mfbaa * (vvx + c1o2) + (mfaaa + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaaa = m0;
			mfbaa = m1;
			mfcaa = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcba * c1o2 + mfbba * (vvx - c1o2) + (mfaba + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcba - c2o1* mfbba *  vvx + mfaba                  * (c1o1 - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcba * c1o2 + mfbba * (vvx + c1o2) + (mfaba + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaba = m0;
			mfbba = m1;
			mfcba = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcca * c1o2 + mfbca * (vvx - c1o2) + (mfaca + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcca - c2o1* mfbca *  vvx + mfaca                   * (c1o1 - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcca * c1o2 + mfbca * (vvx + c1o2) + (mfaca + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaca = m0;
			mfbca = m1;
			mfcca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcab * c1o2 + mfbab * (vvx - c1o2) + (mfaab + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcab - c2o1* mfbab *  vvx + mfaab                  * (c1o1 - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcab * c1o2 + mfbab * (vvx + c1o2) + (mfaab + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaab = m0;
			mfbab = m1;
			mfcab = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 = mfcbb * c1o2 + mfbbb * (vvx - c1o2) + (mfabb + c4o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcbb - c2o1* mfbbb *  vvx + mfabb                  * (c1o1 - vx2) - c4o9 * oMdrho * vx2;
			m2 = mfcbb * c1o2 + mfbbb * (vvx + c1o2) + (mfabb + c4o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfabb = m0;
			mfbbb = m1;
			mfcbb = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 = mfccb * c1o2 + mfbcb * (vvx - c1o2) + (mfacb + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfccb - c2o1* mfbcb *  vvx + mfacb                  * (c1o1 - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfccb * c1o2 + mfbcb * (vvx + c1o2) + (mfacb + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfacb = m0;
			mfbcb = m1;
			mfccb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcac * c1o2 + mfbac * (vvx - c1o2) + (mfaac + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcac - c2o1* mfbac *  vvx + mfaac                   * (c1o1 - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcac * c1o2 + mfbac * (vvx + c1o2) + (mfaac + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaac = m0;
			mfbac = m1;
			mfcac = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 = mfcbc * c1o2 + mfbbc * (vvx - c1o2) + (mfabc + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcbc - c2o1* mfbbc *  vvx + mfabc                  * (c1o1 - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcbc * c1o2 + mfbbc * (vvx + c1o2) + (mfabc + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfabc = m0;
			mfbbc = m1;
			mfcbc = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfbcc * (vvx - c1o2) + (mfacc + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfccc - c2o1* mfbcc *  vvx + mfacc                   * (c1o1 - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfccc * c1o2 + mfbcc * (vvx + c1o2) + (mfacc + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfacc = m0;
			mfbcc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			(D27.f[DIR_P00])[k] = mfabb;
			(D27.f[DIR_M00])[kw] = mfcbb;
			(D27.f[DIR_0P0])[k] = mfbab;
			(D27.f[DIR_0M0])[ks] = mfbcb;
			(D27.f[DIR_00P])[k] = mfbba;
			(D27.f[DIR_00M])[kb] = mfbbc;
			(D27.f[DIR_PP0])[k] = mfaab;
			(D27.f[DIR_MM0])[ksw] = mfccb;
			(D27.f[DIR_PM0])[ks] = mfacb;
			(D27.f[DIR_MP0])[kw] = mfcab;
			(D27.f[DIR_P0P])[k] = mfaba;
			(D27.f[DIR_M0M])[kbw] = mfcbc;
			(D27.f[DIR_P0M])[kb] = mfabc;
			(D27.f[DIR_M0P])[kw] = mfcba;
			(D27.f[DIR_0PP])[k] = mfbaa;
			(D27.f[DIR_0MM])[kbs] = mfbcc;
			(D27.f[DIR_0PM])[kb] = mfbac;
			(D27.f[DIR_0MP])[ks] = mfbca;
			(D27.f[DIR_000])[k] = mfbbb;
			(D27.f[DIR_PPP])[k] = mfaaa;
			(D27.f[DIR_PMP])[ks] = mfaca;
			(D27.f[DIR_PPM])[kb] = mfaac;
			(D27.f[DIR_PMM])[kbs] = mfacc;
			(D27.f[DIR_MPP])[kw] = mfcaa;
			(D27.f[DIR_MMP])[ksw] = mfcca;
			(D27.f[DIR_MPM])[kbw] = mfcac;
			(D27.f[DIR_MMM])[kbsw] = mfccc;
			////////////////////////////////////////////////////////////////////////////////////

		}
	}
}