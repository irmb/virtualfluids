#include "hip/hip_runtime.h"
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;
#include "math.h"

extern "C" __global__ void LB_Kernel_AD_Incomp_7(real diffusivity,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* DDStart,
	real* DD7,
	int size_Mat,
	bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if ((BC != GEO_SOLID) && (BC != GEO_VOID))
		{
			Distributions27 D;
			if (EvenOrOdd == true)
			{
				D.f[E] = &DDStart[E   *size_Mat];
				D.f[W] = &DDStart[W   *size_Mat];
				D.f[N] = &DDStart[N   *size_Mat];
				D.f[S] = &DDStart[S   *size_Mat];
				D.f[T] = &DDStart[T   *size_Mat];
				D.f[B] = &DDStart[B   *size_Mat];
				D.f[NE] = &DDStart[NE  *size_Mat];
				D.f[SW] = &DDStart[SW  *size_Mat];
				D.f[SE] = &DDStart[SE  *size_Mat];
				D.f[NW] = &DDStart[NW  *size_Mat];
				D.f[TE] = &DDStart[TE  *size_Mat];
				D.f[BW] = &DDStart[BW  *size_Mat];
				D.f[BE] = &DDStart[BE  *size_Mat];
				D.f[TW] = &DDStart[TW  *size_Mat];
				D.f[TN] = &DDStart[TN  *size_Mat];
				D.f[BS] = &DDStart[BS  *size_Mat];
				D.f[BN] = &DDStart[BN  *size_Mat];
				D.f[TS] = &DDStart[TS  *size_Mat];
				D.f[dirREST] = &DDStart[dirREST*size_Mat];
				D.f[TNE] = &DDStart[TNE *size_Mat];
				D.f[TSW] = &DDStart[TSW *size_Mat];
				D.f[TSE] = &DDStart[TSE *size_Mat];
				D.f[TNW] = &DDStart[TNW *size_Mat];
				D.f[BNE] = &DDStart[BNE *size_Mat];
				D.f[BSW] = &DDStart[BSW *size_Mat];
				D.f[BSE] = &DDStart[BSE *size_Mat];
				D.f[BNW] = &DDStart[BNW *size_Mat];
			}
			else
			{
				D.f[W] = &DDStart[E   *size_Mat];
				D.f[E] = &DDStart[W   *size_Mat];
				D.f[S] = &DDStart[N   *size_Mat];
				D.f[N] = &DDStart[S   *size_Mat];
				D.f[B] = &DDStart[T   *size_Mat];
				D.f[T] = &DDStart[B   *size_Mat];
				D.f[SW] = &DDStart[NE  *size_Mat];
				D.f[NE] = &DDStart[SW  *size_Mat];
				D.f[NW] = &DDStart[SE  *size_Mat];
				D.f[SE] = &DDStart[NW  *size_Mat];
				D.f[BW] = &DDStart[TE  *size_Mat];
				D.f[TE] = &DDStart[BW  *size_Mat];
				D.f[TW] = &DDStart[BE  *size_Mat];
				D.f[BE] = &DDStart[TW  *size_Mat];
				D.f[BS] = &DDStart[TN  *size_Mat];
				D.f[TN] = &DDStart[BS  *size_Mat];
				D.f[TS] = &DDStart[BN  *size_Mat];
				D.f[BN] = &DDStart[TS  *size_Mat];
				D.f[dirREST] = &DDStart[dirREST*size_Mat];
				D.f[BSW] = &DDStart[TNE *size_Mat];
				D.f[BNE] = &DDStart[TSW *size_Mat];
				D.f[BNW] = &DDStart[TSE *size_Mat];
				D.f[BSE] = &DDStart[TNW *size_Mat];
				D.f[TSW] = &DDStart[BNE *size_Mat];
				D.f[TNE] = &DDStart[BSW *size_Mat];
				D.f[TNW] = &DDStart[BSE *size_Mat];
				D.f[TSE] = &DDStart[BNW *size_Mat];
			}

			Distributions7 D7;
			if (EvenOrOdd == true)
			{
				D7.f[0] = &DD7[0 * size_Mat];
				D7.f[1] = &DD7[1 * size_Mat];
				D7.f[2] = &DD7[2 * size_Mat];
				D7.f[3] = &DD7[3 * size_Mat];
				D7.f[4] = &DD7[4 * size_Mat];
				D7.f[5] = &DD7[5 * size_Mat];
				D7.f[6] = &DD7[6 * size_Mat];
			}
			else
			{
				D7.f[0] = &DD7[0 * size_Mat];
				D7.f[2] = &DD7[1 * size_Mat];
				D7.f[1] = &DD7[2 * size_Mat];
				D7.f[4] = &DD7[3 * size_Mat];
				D7.f[3] = &DD7[4 * size_Mat];
				D7.f[6] = &DD7[5 * size_Mat];
				D7.f[5] = &DD7[6 * size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			unsigned int kw = neighborX[k];
			unsigned int ks = neighborY[k];
			unsigned int kb = neighborZ[k];
			unsigned int ksw = neighborY[kw];
			unsigned int kbw = neighborZ[kw];
			unsigned int kbs = neighborZ[ks];
			unsigned int kbsw = neighborZ[ksw];
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real fW = (D.f[E])[k];//ke
			real fE = (D.f[W])[kw];
			real fS = (D.f[N])[k];//kn
			real fN = (D.f[S])[ks];
			real fB = (D.f[T])[k];//kt
			real fT = (D.f[B])[kb];
			real fSW = (D.f[NE])[k];//kne
			real fNE = (D.f[SW])[ksw];
			real fNW = (D.f[SE])[ks];//kse
			real fSE = (D.f[NW])[kw];//knw
			real fBW = (D.f[TE])[k];//kte
			real fTE = (D.f[BW])[kbw];
			real fTW = (D.f[BE])[kb];//kbe
			real fBE = (D.f[TW])[kw];//ktw
			real fBS = (D.f[TN])[k];//ktn
			real fTN = (D.f[BS])[kbs];
			real fTS = (D.f[BN])[kb];//kbn
			real fBN = (D.f[TS])[ks];//kts
			//real fZERO = (D.f[dirREST])[k];//kzero
			real fBSW = (D.f[TNE])[k];//ktne
			real fBNE = (D.f[TSW])[ksw];//ktsw
			real fBNW = (D.f[TSE])[ks];//ktse
			real fBSE = (D.f[TNW])[kw];//ktnw
			real fTSW = (D.f[BNE])[kb];//kbne
			real fTNE = (D.f[BSW])[kbsw];
			real fTNW = (D.f[BSE])[kbs];//kbse
			real fTSE = (D.f[BNW])[kbw];//kbnw
										   //real fE    =  (D.f[E   ])[k  ];//ke
										   //real fW    =  (D.f[W   ])[kw ];
										   //real fN    =  (D.f[N   ])[k  ];//kn
										   //real fS    =  (D.f[S   ])[ks ];
										   //real fT    =  (D.f[T   ])[k  ];//kt
										   //real fB    =  (D.f[B   ])[kb ];
										   //real fNE   =  (D.f[NE  ])[k  ];//kne
										   //real fSW   =  (D.f[SW  ])[ksw];
										   //real fSE   =  (D.f[SE  ])[ks ];//kse
										   //real fNW   =  (D.f[NW  ])[kw ];//knw
										   //real fTE   =  (D.f[TE  ])[k  ];//kte
										   //real fBW   =  (D.f[BW  ])[kbw];
										   //real fBE   =  (D.f[BE  ])[kb ];//kbe
										   //real fTW   =  (D.f[TW  ])[kw ];//ktw
										   //real fTN   =  (D.f[TN  ])[k  ];//ktn
										   //real fBS   =  (D.f[BS  ])[kbs];
										   //real fBN   =  (D.f[BN  ])[kb ];//kbn
										   //real fTS   =  (D.f[TS  ])[ks ];//kts
										   //real fZERO =  (D.f[dirREST])[k  ];//kzero
										   //real fTNE   = (D.f[TNE ])[k  ];//ktne
										   //real fTSW   = (D.f[TSW ])[ksw];//ktsw
										   //real fTSE   = (D.f[TSE ])[ks ];//ktse
										   //real fTNW   = (D.f[TNW ])[kw ];//ktnw
										   //real fBNE   = (D.f[BNE ])[kb ];//kbne
										   //real fBSW   = (D.f[BSW ])[kbsw];
										   //real fBSE   = (D.f[BSE ])[kbs];//kbse
										   //real fBNW   = (D.f[BNW ])[kbw];//kbnw
										   ////////////////////////////////////////////////////////////////////////////////
			real f7ZERO = (D7.f[0])[k];
			real f7E = (D7.f[1])[k];
			real f7W = (D7.f[2])[kw];
			real f7N = (D7.f[3])[k];
			real f7S = (D7.f[4])[ks];
			real f7T = (D7.f[5])[k];
			real f7B = (D7.f[6])[kb];
			////////////////////////////////////////////////////////////////////////////////
			real vx = ((fTNE - fBSW) + (fBNE - fTSW) + (fTSE - fBNW) + (fBSE - fTNW) + (fNE - fSW) + (fSE - fNW) + (fTE - fBW) + (fBE - fTW) + (fE - fW));
			real vy = ((fTNE - fBSW) + (fBNE - fTSW) + (fBNW - fTSE) + (fTNW - fBSE) + (fNE - fSW) + (fNW - fSE) + (fTN - fBS) + (fBN - fTS) + (fN - fS));
			real vz = ((fTNE - fBSW) + (fTSW - fBNE) + (fTSE - fBNW) + (fTNW - fBSE) + (fTE - fBW) + (fTW - fBE) + (fTN - fBS) + (fTS - fBN) + (fT - fB));
			////d�rrrrrty !!!!!!!!!!!!!
			//      real vx     =  ten * ((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
			//      real vy     =  ten * ((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
			//      real vz     =  ten * ((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
			////////////////////////////////////////////////////////////////////////////////
			//real ux_sq = vx * vx;
			//real uy_sq = vy * vy;
			//real uz_sq = vz * vz;
			////////////////////////////////////////////////////////////////////////////////
			//BGK
			//real omegaD     = -three + sqrt(three); !!!!!!!!!!!!!!Achtung!!!!!!!!!!!!!!!!!! anderes Vorzeichen als in den Randbedingungen
			//real Lam         = -(c1o2+one/omegaD);
			//real nue_d       = Lam/three;
			//real ae          = diffusivity/nue_d - one;

			//real ConcD       = f7ZERO+f7E+f7W+f7N+f7S+f7T+f7B;

			//(D7.f[0])[k  ] = f7ZERO*(one+omegaD)-omegaD*ConcD*(c1o3*(ae*(-three))-(ux_sq+uy_sq+uz_sq));
			//(D7.f[2])[kw ] = f7E   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)+vx*c1o2);
			//(D7.f[1])[k  ] = f7W   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)-vx*c1o2);
			//(D7.f[4])[ks ] = f7N   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)+vy*c1o2);
			//(D7.f[3])[k  ] = f7S   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)-vy*c1o2);
			//(D7.f[6])[kb ] = f7T   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)+vz*c1o2);
			//(D7.f[5])[k  ] = f7B   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)-vz*c1o2);

			////////////////////////////////////////////////////////////////////////////////
			//TRT  Yoshida Kernel - based on Ying
			//real cs2 = c1o4;
			real Lam = diffusivity*c4o1;//diffusivity/(one)/cs2;
			real omegaD = -c1o1 / (Lam + c1o2);
			//real ae = c0o1;
			////////////////////////////////////////////////////////////////////////////////
			real ConcD = f7ZERO + f7E + f7W + f7N + f7S + f7T + f7B;

			real Mom000 = f7ZERO + f7W + f7E + f7N + f7S + f7T + f7B; //1
			real Mom100 = f7E - f7W;
			real Mom010 = f7N - f7S;
			real Mom001 = f7T - f7B;
			real Mom222 = c6o1*f7ZERO - f7W - f7E - f7N - f7S - f7T - f7B;
			real Mom200 = c2o1*f7W + c2o1*f7E - f7N - f7S - f7T - f7B;
			real Mom022 = f7N + f7S - f7T - f7B;

			real Meq000 = ConcD;
			real Meq100 = ConcD*vx;
			real Meq010 = ConcD*vy;
			real Meq001 = ConcD*vz;
			real Meq222 = c3o4*ConcD;
			real Meq200 = c0o1;
			real Meq022 = c0o1;

			// relaxation TRT Yoshida

			// odd 
			Mom100 = omegaD * (Mom100 - Meq100);
			Mom010 = omegaD * (Mom010 - Meq010);
			Mom001 = omegaD * (Mom001 - Meq001);

			// even
			Mom000 = -c1o1*(Mom000 - Meq000);
			Mom222 = -c1o1*(Mom222 - Meq222);
			Mom200 = -c1o1*(Mom200 - Meq200);
			Mom022 = -c1o1*(Mom022 - Meq022);

			//Back transformation to distributions
			f7ZERO = f7ZERO + c1o7*Mom000 + c1o7*Mom222;                                                  //1
			f7E = f7E + c1o7*Mom000 + c1o2*Mom100 - c1o6*c1o7*Mom222 + c1o6*Mom200;                 //2
			f7W = f7W + c1o7*Mom000 - c1o2*Mom100 - c1o6*c1o7*Mom222 + c1o6*Mom200;                 //3
			f7N = f7N + c1o7*Mom000 + c1o2*Mom010 - c1o6*c1o7*Mom222 - c1o12*Mom200 + c1o4 *Mom022; //4
			f7S = f7S + c1o7*Mom000 - c1o2*Mom010 - c1o6*c1o7*Mom222 - c1o12*Mom200 + c1o4 *Mom022; //5
			f7T = f7T + c1o7*Mom000 + c1o2*Mom001 - c1o6*c1o7*Mom222 - c1o12*Mom200 - c1o4 *Mom022; //6
			f7B = f7B + c1o7*Mom000 - c1o2*Mom001 - c1o6*c1o7*Mom222 - c1o12*Mom200 - c1o4 *Mom022; //7

			(D7.f[0])[k] = f7ZERO;
			(D7.f[2])[kw] = f7E;
			(D7.f[1])[k] = f7W;
			(D7.f[4])[ks] = f7N;
			(D7.f[3])[k] = f7S;
			(D7.f[6])[kb] = f7T;
			(D7.f[5])[k] = f7B;
		}
	}
}