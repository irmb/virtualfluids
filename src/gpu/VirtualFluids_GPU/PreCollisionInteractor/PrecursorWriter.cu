#include "hip/hip_runtime.h"
#include "PrecursorWriter.h"
#include "basics/writer/WbWriterVtkXmlImageBinary.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda/CudaGrid.h>
#include "Kernel/Utilities/DistributionHelper.cuh"

#include <Core/StringUtilities/StringUtil.h>

#include "Parameter/Parameter.h"
#include "DataStructureInitializer/GridProvider.h"
#include "GPU/CudaMemoryManager.h"

using namespace vf::lbm::dir;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//TODO check everything for multiple level
void index1d(int& idx, int y, int z, int ny, int nz)
{
    idx = y+ny*z;
}

void index2d(int idx, int& y, int& z, int ny, int nz)
{
    z = idx/ny;
    y = idx-ny*z;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__inline__ __device__ __host__ void getPointersToDistributionSubset9(DistributionReferencesSubset9 &dist, real *distributionArray, const uint numberOfNodes)
{
    dist.f[0   ]   = &distributionArray[0   *numberOfNodes];
    dist.f[1   ]   = &distributionArray[1   *numberOfNodes];
    dist.f[2   ]   = &distributionArray[2   *numberOfNodes];
    dist.f[3   ]   = &distributionArray[3   *numberOfNodes];
    dist.f[4   ]   = &distributionArray[4   *numberOfNodes];
    dist.f[5   ]   = &distributionArray[5   *numberOfNodes];
    dist.f[6   ]   = &distributionArray[6   *numberOfNodes];
    dist.f[7   ]   = &distributionArray[7   *numberOfNodes];
    dist.f[8   ]   = &distributionArray[8   *numberOfNodes];
    
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__inline__ __device__ __host__ DistributionReferencesSubset9 getDistributionReferencesSubset9(real* distributionSubset, unsigned int numberOfNodes)
{
    DistributionReferencesSubset9 distribution_references;
    getPointersToDistributionSubset9(distribution_references, distributionSubset, numberOfNodes);
    return distribution_references;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void fillArrayVelocities(uint nNodes, uint* indices, 
                                    real *precursorVx,
                                    real *precursorVy, 
                                    real *precursorVz,  
                                    real *vx,
                                    real *vy,
                                    real *vz,
                                    real velocityRatio)


{
    const uint node = vf::gpu::getNodeIndex();

    if(node>=nNodes) return;

    precursorVx[node] = vx[indices[node]]*velocityRatio;
    precursorVy[node] = vy[indices[node]]*velocityRatio;
    precursorVz[node] = vz[indices[node]]*velocityRatio;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void fillArrayDistributions( uint nNodes, uint* indices, 
                                        real* precursorDistributions,
                                        real* distributions,
                                        uint* neighborX, uint* neighborY, uint* neighborZ,
                                        bool isEvenTimestep,
                                        unsigned long numberOfLBnodes)
{
    const uint node = vf::gpu::getNodeIndex();

    if(node>=nNodes) return;

    Distributions27 dist = vf::gpu::getDistributionReferences27(distributions, numberOfLBnodes, isEvenTimestep);

    DistributionSubset9 distPrecursor = getDistributionReferencesSubset9(distributions, nNodes);    
    
    ////////////////////////////////////////////////////////////////////////////////
    // ! - Set neighbor indices (necessary for indirect addressing)
    uint k_000 = indices[node];
    // uint k_M00 = neighborX[k_000];
    uint k_0M0 = neighborY[k_000];
    uint k_00M = neighborZ[k_000];
    // uint k_MM0 = neighborY[k_M00];
    // uint k_M0M = neighborZ[k_M00];
    uint k_0MM = neighborZ[k_0M0];
    // uint k_MMM = neighborZ[k_MM0];

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Get local distributions in PX directions
    //!
    (distPrecursor.f[0])[node] = (dist.f[DIR_P00])[k_000];
    (distPrecursor.f[1])[node] = (dist.f[DIR_PP0])[k_000];
    (distPrecursor.f[2])[node] = (dist.f[DIR_PM0])[k_0M0];
    (distPrecursor.f[3])[node] = (dist.f[DIR_P0P])[k_000];
    (distPrecursor.f[4])[node] = (dist.f[DIR_P0M])[k_00M];
    (distPrecursor.f[5])[node] = (dist.f[DIR_PPP])[k_000];
    (distPrecursor.f[6])[node] = (dist.f[DIR_PMP])[k_0M0];
    (distPrecursor.f[7])[node] = (dist.f[DIR_PPM])[k_00M];
    (distPrecursor.f[8])[node] = (dist.f[DIR_PMM])[k_0MM];
    
    if(node==1)
        printf("thread %i, pos %i, f0 %f \n", node, indices[node], (distPrecursor.f[0])[node]);
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void PrecursorWriter::init(Parameter* para, GridProvider* gridProvider, CudaMemoryManager* cudaManager)
{
    precursorStructs.resize(para->getMaxLevel()+1);
    for(int level=0; level<=para->getMaxLevel(); level++)
    {

        real dx = abs(para->getParH(level)->coordinateX[1]-para->getParH(level)->coordinateX[para->getParH(level)->neighborX[1]]);
        int maxPoints = (int((yMax-yMin)/dx)+1)* (int((zMax-zMin)/dx)+1);

        real lowestY, lowestZ, highestY, highestZ;

        lowestY = para->getParH(level)->coordinateY[para->getParH(level)->numberOfNodes-1];
        highestY = para->getParH(level)->coordinateY[1];        
        
        lowestZ = para->getParH(level)->coordinateZ[para->getParH(level)->numberOfNodes-1];
        highestZ = para->getParH(level)->coordinateZ[1];

        std::vector<uint> indicesOnGrid;
        std::vector<int> indicesOnPlane;
        std::vector<real> coordY, coordZ;

        for(uint j=1; j<para->getParH(level)->numberOfNodes; j++ )
        {
            real pointCoordX = para->getParH(level)->coordinateX[j];
            real pointCoordY = para->getParH(level)->coordinateY[j];
            real pointCoordZ = para->getParH(level)->coordinateZ[j];
            if( pointCoordX < (dx+xPos) && pointCoordX >= xPos &&
                pointCoordY<=yMax && pointCoordY>=yMin && 
                pointCoordZ<=zMax && pointCoordZ>=zMin)
            {
                highestY = max(highestY, pointCoordY);
                highestZ = max(highestZ, pointCoordZ);

                lowestY = min(lowestY, pointCoordY);
                lowestZ = min(lowestZ, pointCoordZ);
                indicesOnGrid.push_back(j);    
                coordY.push_back(pointCoordY);            
                coordZ.push_back(pointCoordZ);            
            }
        }
        assert("PrecursorWriter did not find any points on the grid"&& indicesOnGrid.size()==0);
        int ny = int((highestY-lowestY)/dx)+1;
        int nz = int((highestZ-lowestZ)/dx)+1;
        printf("ny %d nz %d \n", ny, nz);
        for(uint i=0;i<indicesOnGrid.size(); i++)
        {
                int idxY = int((coordY[i]-lowestY)/dx);
                int idxZ = int((coordZ[i]-lowestZ)/dx);
                int idx;
                index1d(idx, idxY, idxZ, ny, nz);
                indicesOnPlane.push_back(idx);
                // printf("idx %d, idy %d, idz %d, ny %d, nz %d\n", idx, idxY, idxZ, ny, nz);
        }

        precursorStructs[level] = SPtr<PrecursorStruct>(new PrecursorStruct);
        precursorStructs[level]->nPoints = (uint)indicesOnGrid.size();
        precursorStructs[level]->indicesOnPlane = (int*) malloc(precursorStructs[level]->nPoints*sizeof(int));
        precursorStructs[level]->spacing = makeUbTuple(dx, dx, tSave*para->getTimeRatio());
        precursorStructs[level]->origin = makeUbTuple(lowestY, lowestZ);
        precursorStructs[level]->extent = makeUbTuple(0, ny-1, 0, nz-1);
        precursorStructs[level]->nPointsInPlane = ny*nz;
        precursorStructs[level]->timestepsPerFile = min(para->getlimitOfNodesForVTK()/(ny*nz), maxtimestepsPerFile);
        precursorStructs[level]->filesWritten = 0;
        precursorStructs[level]->timestepsBuffered = 0;

        printf("points %zu points on plane %zu \n",  indicesOnGrid.size(),  indicesOnPlane.size());

        cudaManager->cudaAllocPrecursorWriter(this, level);
    
        std::copy(indicesOnGrid.begin(), indicesOnGrid.end(), precursorStructs[level]->indicesH);
        std::copy(indicesOnPlane.begin(), indicesOnPlane.end(), precursorStructs[level]->indicesOnPlane);

        cudaManager->cudaCopyPrecursorWriterIndicesHtoD(this, level);
    }
}


void PrecursorWriter::interact(Parameter* para, CudaMemoryManager* cudaManager, int level, uint t)
{
    if(t>tStartOut ? ((t-tStartOut) % tSave)==0 : false)
    {
        SPtr<PrecursorStruct> precursorStruct = precursorStructs[level];
        vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(para->getParH(level)->numberofthreads, precursorStruct->nPoints);

        if(this->outputVariable==OutputVariable::Velocities)
        {
            fillArrayVelocities<<<grid.grid, grid.threads>>>(   precursorStruct->nPoints, precursorStruct->indicesD, 
                                                                precursorStruct->vxD, precursorStruct->vyD, precursorStruct->vzD, 
                                                                para->getParD(level)->velocityX, para->getParD(level)->velocityY, para->getParD(level)->velocityZ,
                                                                para->getVelocityRatio());
            getLastCudaError("In PrecursorWriter::interact fillArrayVelocities execution failed");
        }
        else if(this->outputVariable==OutputVariable::Distributions)
        {
            fillArrayDistributions<<<grid.grid, grid.threads>>>(precursorStruct->nPoints, precursorStruct->indicesD, 
                                                                precursorStruct->distD.f[0],
                                                                para->getParD(level)->distributions.f[0],
                                                                para->getParD(level)->neighborX, para->getParD(level)->neighborY, para->getParD(level)->neighborZ,
                                                                para->getEvenOrOdd(level), para->getParD(level)->numberOfNodes);
            getLastCudaError("In PrecursorWriter::interact fillArrayDistributions execution failed");
        }

        cudaManager->cudaCopyPrecursorWriterOutputVariablesDtoH(this, level);
        
        DistributionSubset9 distPrecursor = getDistributionReferencesSubset9(precursorStruct->distH.f[0], precursorStruct->nPoints*precursorStruct->timestepsBuffered);
        uint node = 1;
        int idx = node+t*precursorStruct->nPoints;
        printf("host %i, pos %i, f0 %f \n", node, precursorStruct->indicesH[node], (distPrecursor.f[0])[idx]);

        precursorStruct->timestepsBuffered++;

        if(precursorStruct->timestepsBuffered >= precursorStruct->timestepsPerFile)
            this->write(para, level);
    }
}


void PrecursorWriter::free(Parameter* para, CudaMemoryManager* cudaManager)
{
    for(int level=0; level<=para->getMaxLevel(); level++)
    {
        if(getPrecursorStruct(level)->timestepsBuffered>0)
            write(para, level);

        cudaManager->cudaFreePrecursorWriter(this, level);
    }
}


void PrecursorWriter::write(Parameter* para, int level)
{
    SPtr<PrecursorStruct> precursorStruct = this->getPrecursorStruct(level);
    std::string fname = this->makeFileName(fileName, level, para->getMyProcessID(), precursorStruct->filesWritten) + getWriter()->getFileExtension();
    std::string wholeName = outputPath + "/" + fname;

    uint nPointsInPlane = precursorStruct->nPointsInPlane;

    int startTime = precursorStruct->filesWritten*precursorStruct->timestepsPerFile;

    // printf("points in plane %d, total timesteps %d, ntimesteps %d \n", nPointsInPlane, nTotalTimesteps, nTimesteps);

    UbTupleInt6 extent = makeUbTuple(   val<1>(precursorStruct->extent),    val<2>(precursorStruct->extent), 
                                        val<3>(precursorStruct->extent),    val<4>(precursorStruct->extent), 
                                        startTime,                          startTime+(int)precursorStruct->timestepsBuffered-1);

    UbTupleFloat3 origin = makeUbTuple( val<1>(precursorStruct->origin), val<2>(precursorStruct->origin), 0.f);

    std::vector<std::vector<double>> nodedata;

    if(this->outputVariable==OutputVariable::Velocities)
    {
        std::vector<double> vxDouble(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                            vyDouble(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                            vzDouble(nPointsInPlane*precursorStruct->timestepsBuffered, NAN);

        for( uint timestep=0; timestep<precursorStruct->timestepsBuffered; timestep++)
        {
            // printf("offset %d npoints %d buf %d, max%d\n",timestep, precursorStruct->nPoints, precursorStruct->timestepsBuffered, precursorStruct->timestepsPerFile);
            for (uint pos = 0; pos < precursorStruct->nPoints; pos++)
            {
                int indexOnPlane = precursorStruct->indicesOnPlane[pos]+timestep*nPointsInPlane;
                int idx = pos+timestep*precursorStruct->nPoints;
                // printf("timestep %i, pos %i, iOP %i \n", timestep, pos, indexOnPlane);
                // printf("vx %f, vy %f, vz%f nodedata x %f\n", vx[level][timestep][pos], vy[level][timestep][pos], vz[level][timestep][pos], vxDouble[indexOnPlane]);
                vxDouble[indexOnPlane] = double(precursorStruct->vxH[idx]);
                vyDouble[indexOnPlane] = double(precursorStruct->vyH[idx]);
                vzDouble[indexOnPlane] = double(precursorStruct->vzH[idx]);
            }
        }
        nodedata = {vxDouble, vyDouble, vzDouble};
    }
    else if(this->outputVariable==OutputVariable::Distributions)
    {
                std::vector<double> f0Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                                    f1Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                                    f2Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                                    f3Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                                    f4Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                                    f5Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                                    f6Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                                    f7Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN), 
                                    f8Double(nPointsInPlane*precursorStruct->timestepsBuffered, NAN);

        DistributionSubset9 distPrecursor = getDistributionReferencesSubset9(precursorStruct->distH.f[0], precursorStruct->nPoints*precursorStruct->timestepsBuffered);

        for( uint timestep=0; timestep<precursorStruct->timestepsBuffered; timestep++)
        {
            printf("offset %d npoints %d buf %d, max%d\n",timestep, precursorStruct->nPoints, precursorStruct->timestepsBuffered, precursorStruct->timestepsPerFile);
            for (uint pos = 0; pos < precursorStruct->nPoints; pos++)
            {
                int indexOnPlane = precursorStruct->indicesOnPlane[pos]+timestep*nPointsInPlane;
                int idx = pos+timestep*precursorStruct->nPoints;
                printf("timestep %i, pos %i, iOP %i \n", timestep, pos, indexOnPlane);
                printf("f0 %f\n", double((distPrecursor.f[0])[idx]));
                f0Double[indexOnPlane] = double((distPrecursor.f[0])[idx]);
                f1Double[indexOnPlane] = double((distPrecursor.f[1])[idx]);
                f2Double[indexOnPlane] = double((distPrecursor.f[2])[idx]);
                f3Double[indexOnPlane] = double((distPrecursor.f[3])[idx]);
                f4Double[indexOnPlane] = double((distPrecursor.f[4])[idx]);
                f5Double[indexOnPlane] = double((distPrecursor.f[5])[idx]);
                f6Double[indexOnPlane] = double((distPrecursor.f[6])[idx]);
                f7Double[indexOnPlane] = double((distPrecursor.f[7])[idx]);
                f8Double[indexOnPlane] = double((distPrecursor.f[8])[idx]);
            }
        }
        nodedata = {f0Double, f1Double, f2Double, f3Double, f4Double, f5Double, f6Double, f7Double, f8Double};
    }

    precursorStruct->timestepsBuffered = 0;

    std::vector<std::vector<double>> celldata;
    getWriter()->writeData(wholeName, nodedatanames, celldatanames, nodedata, celldata, extent, origin, precursorStruct->spacing, extent);
    precursorStruct->filesWritten++;
}

std::string PrecursorWriter::makeFileName(std::string fileName, int level, int id, uint filesWritten)
{
    return fileName + "_lev_" + StringUtil::toString<int>(level)
                    + "_ID_" + StringUtil::toString<int>(id)
                    + "_File_" + StringUtil::toString<int>(filesWritten);
}