#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __         
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |        
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |        
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |        
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____    
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|   
//      \    \  |    |   ________________________________________________________________    
//       \    \ |    |  |  ______________________________________________________________|   
//        \    \|    |  |  |         __          __     __     __     ______      _______    
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)   
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______    
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/   
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can 
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of 
//  the License, or (at your option) any later version.
//  
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT 
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or 
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License 
//  for more details.
//  
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file Probe.h
//! \author Henry Korb, Henrik Asmuth
//=======================================================================================

#include "Probe.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "VirtualFluids_GPU/GPU/GeometryUtils.h"
#include <lbm/constants/NumericConstants.h>
#include "basics/writer/WbWriterVtkXmlBinary.h"
#include <Core/StringUtilities/StringUtil.h>

#include "Parameter/Parameter.h"
#include "DataStructureInitializer/GridProvider.h"
#include "GPU/CudaMemoryManager.h"

using namespace vf::lbm::constant;

__host__ __device__ int calcArrayIndex(int node, int nNodes, int timestep, int nTimesteps, int array)
{
    return node+nNodes*(timestep+nTimesteps*array);
}

uint calcOldTimestep(uint currentTimestep, uint lastTimestepInOldSeries)
{
    return currentTimestep > 0 ? currentTimestep - 1 : lastTimestepInOldSeries; 
}

__device__ void calculatePointwiseQuantities(
    uint oldTimestepInTimeseries,
    uint timestepInTimeseries,
    uint timestepInAverage,
    uint nTimesteps,
    real* quantityArray,
    bool* quantities,
    uint* quantityArrayOffsets,
    uint nPoints,
    uint node,
    real vx,
    real vy,
    real vz,
    real rho)
{
    //"https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Welford's_online_algorithm"
    // also has extensions for higher order and covariances
    int n = timestepInAverage+1;
    real inv_n = 1/real(n);


    if(quantities[int(Statistic::Instantaneous)])
    {
        uint arrOff = quantityArrayOffsets[int(Statistic::Instantaneous)];
        quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+0)] = vx;
        quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+1)] = vy;
        quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+2)] = vz;
        quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+3)] = rho;
    }


    if(quantities[int(Statistic::Means)])
    {
        
        uint arrOff = quantityArrayOffsets[int(Statistic::Means)];
        real vx_m_old  = quantityArray[calcArrayIndex(node, nPoints, oldTimestepInTimeseries, nTimesteps, arrOff+0)];
        real vy_m_old  = quantityArray[calcArrayIndex(node, nPoints, oldTimestepInTimeseries, nTimesteps, arrOff+1)];
        real vz_m_old  = quantityArray[calcArrayIndex(node, nPoints, oldTimestepInTimeseries, nTimesteps, arrOff+2)];
        real rho_m_old = quantityArray[calcArrayIndex(node, nPoints, oldTimestepInTimeseries, nTimesteps, arrOff+3)];

        real vx_m_new  = ( (n-1)*vx_m_old + vx  )*inv_n;
        real vy_m_new  = ( (n-1)*vy_m_old + vy  )*inv_n;
        real vz_m_new  = ( (n-1)*vz_m_old + vz  )*inv_n;
        real rho_m_new = ( (n-1)*rho_m_old+ rho )*inv_n;

        quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+0)] = vx_m_new;
        quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+1)] = vy_m_new;
        quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+2)] = vz_m_new;
        quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+3)] = rho_m_new;
    
        if(quantities[int(Statistic::Variances)])
        {
            arrOff = quantityArrayOffsets[int(Statistic::Variances)];

            real vx_var_old  = quantityArray[calcArrayIndex(node, nPoints, oldTimestepInTimeseries, nTimesteps, arrOff+0)];
            real vy_var_old  = quantityArray[calcArrayIndex(node, nPoints, oldTimestepInTimeseries, nTimesteps, arrOff+1)];
            real vz_var_old  = quantityArray[calcArrayIndex(node, nPoints, oldTimestepInTimeseries, nTimesteps, arrOff+2)];
            real rho_var_old = quantityArray[calcArrayIndex(node, nPoints, oldTimestepInTimeseries, nTimesteps, arrOff+3)];

            real vx_var_new  = ( (n-1)*(vx_var_old )+(vx  - vx_m_old )*(vx  - vx_m_new ) )*inv_n;
            real vy_var_new  = ( (n-1)*(vy_var_old )+(vy  - vy_m_old )*(vy  - vy_m_new ) )*inv_n;
            real vz_var_new  = ( (n-1)*(vz_var_old )+(vz  - vz_m_old )*(vz  - vz_m_new ) )*inv_n;
            real rho_var_new = ( (n-1)*(rho_var_old)+(rho - rho_m_old)*(rho - rho_m_new) )*inv_n;

            quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+0)] = vx_var_new;
            quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+1)] = vy_var_new;
            quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+2)] = vz_var_new;
            quantityArray[calcArrayIndex(node, nPoints, timestepInTimeseries, nTimesteps, arrOff+3)] = rho_var_new; 
        }
    }
}

__global__ void calcQuantitiesKernel(   uint* pointIndices,
                                    uint nPoints, uint oldTimestepInTimeseries, uint timestepInTimeseries, uint timestepInAverage, uint nTimesteps,
                                    real* vx, real* vy, real* vz, real* rho,            
                                    uint* neighborX, uint* neighborY, uint* neighborZ,
                                    bool* quantities,
                                    uint* quantityArrayOffsets, real* quantityArray
                                    )
{
    const uint x = threadIdx.x; 
    const uint y = blockIdx.x;
    const uint z = blockIdx.y;

    const uint nx = blockDim.x;
    const uint ny = gridDim.x;

    const uint node = nx*(ny*z + y) + x;

    if(node>=nPoints) return;

    // Get indices of neighbor nodes. 
    // node referring to BSW cell as seen from probe point
    uint k = pointIndices[node];
    real u_interpX, u_interpY, u_interpZ, rho_interp;

    u_interpX = vx[k];
    u_interpY = vy[k];
    u_interpZ = vz[k];
    rho_interp = rho[k];

    calculatePointwiseQuantities(oldTimestepInTimeseries, timestepInTimeseries, timestepInAverage, nTimesteps, quantityArray, quantities, quantityArrayOffsets, nPoints, node, u_interpX, u_interpY, u_interpZ, rho_interp);

}

__global__ void interpAndCalcQuantitiesKernel(   uint* pointIndices,
                                    uint nPoints, uint oldTimestepInTimeseries, uint timestepInTimeseries, uint timestepInAverage, uint nTimesteps,
                                    real* distX, real* distY, real* distZ,
                                    real* vx, real* vy, real* vz, real* rho,            
                                    uint* neighborX, uint* neighborY, uint* neighborZ,
                                    bool* quantities,
                                    uint* quantityArrayOffsets, real* quantityArray
                                )
{
    const uint x = threadIdx.x; 
    const uint y = blockIdx.x;
    const uint z = blockIdx.y;

    const uint nx = blockDim.x;
    const uint ny = gridDim.x;

    const uint node = nx*(ny*z + y) + x;

    if(node>=nPoints) return;

    // Get indices of neighbor nodes. 
    // node referring to BSW cell as seen from probe point
    uint k = pointIndices[node];
    real u_interpX, u_interpY, u_interpZ, rho_interp;

    uint ke, kn, kt, kne, kte, ktn, ktne;
    getNeighborIndicesOfBSW(  k, ke, kn, kt, kne, kte, ktn, ktne, neighborX, neighborY, neighborZ);

    // Trilinear interpolation of macroscopic quantities to probe point
    real dW, dE, dN, dS, dT, dB;
    getInterpolationWeights(dW, dE, dN, dS, dT, dB, distX[node], distY[node], distZ[node]);

    u_interpX  = trilinearInterpolation( dW, dE, dN, dS, dT, dB, k, ke, kn, kt, kne, kte, ktn, ktne, vx );
    u_interpY  = trilinearInterpolation( dW, dE, dN, dS, dT, dB, k, ke, kn, kt, kne, kte, ktn, ktne, vy );
    u_interpZ  = trilinearInterpolation( dW, dE, dN, dS, dT, dB, k, ke, kn, kt, kne, kte, ktn, ktne, vz );
    rho_interp = trilinearInterpolation( dW, dE, dN, dS, dT, dB, k, ke, kn, kt, kne, kte, ktn, ktne, rho );

    calculatePointwiseQuantities(oldTimestepInTimeseries, timestepInTimeseries, timestepInAverage, nTimesteps, quantityArray, quantities, quantityArrayOffsets, nPoints, node, u_interpX, u_interpY, u_interpZ, rho_interp);

}

bool Probe::getHasDeviceQuantityArray(){ return this->hasDeviceQuantityArray; }

real Probe::getNondimensionalConversionFactor(int level){ return c1o1; }

void Probe::init(Parameter* para, GridProvider* gridProvider, CudaMemoryManager* cudaMemoryManager)
{
    using std::placeholders::_1;
    this->velocityRatio      = std::bind(&Parameter::getScaledVelocityRatio,        para, _1); 
    this->densityRatio       = std::bind(&Parameter::getScaledDensityRatio,         para, _1);
    this->forceRatio         = std::bind(&Parameter::getScaledForceRatio,           para, _1);
    this->stressRatio        = std::bind(&Parameter::getScaledStressRatio,          para, _1);
    this->viscosityRatio     = std::bind(&Parameter::getScaledViscosityRatio,       para, _1);
    this->nondimensional     = std::bind(&Probe::getNondimensionalConversionFactor, this, _1);

    probeParams.resize(para->getMaxLevel()+1);

    for(int level=0; level<=para->getMaxLevel(); level++)
    {
        std::vector<int> probeIndices_level;
        std::vector<real> distX_level;
        std::vector<real> distY_level;
        std::vector<real> distZ_level;        
        std::vector<real> pointCoordsX_level;
        std::vector<real> pointCoordsY_level;
        std::vector<real> pointCoordsZ_level;
        
        this->findPoints(para, gridProvider, probeIndices_level, distX_level, distY_level, distZ_level,      
                       pointCoordsX_level, pointCoordsY_level, pointCoordsZ_level,
                       level);
        
        this->addProbeStruct(para, cudaMemoryManager, probeIndices_level, 
                            distX_level, distY_level, distZ_level, 
                            pointCoordsX_level, pointCoordsY_level, pointCoordsZ_level, 
                            level);

        if(this->outputTimeSeries) timeseriesFileNames.push_back(this->writeTimeseriesHeader(para, level));
    }
}

void Probe::addProbeStruct( Parameter* para, CudaMemoryManager* cudaMemoryManager, std::vector<int>& probeIndices,
                            std::vector<real>& distX, std::vector<real>& distY, std::vector<real>& distZ,   
                            std::vector<real>& pointCoordsX, std::vector<real>& pointCoordsY, std::vector<real>& pointCoordsZ,
                            int level)
{
    probeParams[level] = SPtr<ProbeStruct>(new ProbeStruct);
    probeParams[level]->nTimesteps = this->getNumberOfTimestepsInTimeseries(para, level);
    probeParams[level]->nPoints  = uint(pointCoordsX.size()); // Note, need to have both nPoints and nIndices because they differ in PlanarAverage
    probeParams[level]->nIndices = uint(probeIndices.size());

    probeParams[level]->pointCoordsX = (real*)malloc(probeParams[level]->nPoints*sizeof(real));
    probeParams[level]->pointCoordsY = (real*)malloc(probeParams[level]->nPoints*sizeof(real));
    probeParams[level]->pointCoordsZ = (real*)malloc(probeParams[level]->nPoints*sizeof(real));

    std::copy(pointCoordsX.begin(), pointCoordsX.end(), probeParams[level]->pointCoordsX);
    std::copy(pointCoordsY.begin(), pointCoordsY.end(), probeParams[level]->pointCoordsY);
    std::copy(pointCoordsZ.begin(), pointCoordsZ.end(), probeParams[level]->pointCoordsZ);

    // Note, dist only needed for kernels that do interpolate
    if( distX.size()>0 && distY.size()>0 && distZ.size()>0 )
    {
        probeParams[level]->hasDistances=true;
        cudaMemoryManager->cudaAllocProbeDistances(this, level);
        std::copy(distX.begin(), distX.end(), probeParams[level]->distXH);
        std::copy(distY.begin(), distY.end(), probeParams[level]->distYH);
        std::copy(distZ.begin(), distZ.end(), probeParams[level]->distZH);
        cudaMemoryManager->cudaCopyProbeDistancesHtoD(this, level);
    }  
    
    cudaMemoryManager->cudaAllocProbeIndices(this, level);
    std::copy(probeIndices.begin(), probeIndices.end(), probeParams[level]->pointIndicesH);
    cudaMemoryManager->cudaCopyProbeIndicesHtoD(this, level);

    uint arrOffset = 0;

    cudaMemoryManager->cudaAllocProbeQuantitiesAndOffsets(this, level);

    for( int var=0; var<int(Statistic::LAST); var++)
    {
        if(this->quantities[var])
        {
            probeParams[level]->quantitiesH[var] = true;
            probeParams[level]->arrayOffsetsH[var] = arrOffset;
            arrOffset += uint( this->getPostProcessingVariables(static_cast<Statistic>(var)).size() ); 
        }
    }
    
    cudaMemoryManager->cudaCopyProbeQuantitiesAndOffsetsHtoD(this, level);

    probeParams[level]->nArrays = arrOffset;

    cudaMemoryManager->cudaAllocProbeQuantityArray(this, level);

    std::fill_n(probeParams[level]->quantitiesArrayH, probeParams[level]->nArrays*probeParams[level]->nPoints*probeParams[level]->nTimesteps, c0o1);

    if(this->hasDeviceQuantityArray)
        cudaMemoryManager->cudaCopyProbeQuantityArrayHtoD(this, level);

}

void Probe::interact(Parameter* para, CudaMemoryManager* cudaMemoryManager, int level, uint t)
{
    uint t_level = para->getTimeStep(level, t, false);

    SPtr<ProbeStruct> probeStruct = this->getProbeStruct(level);

    //!Skip empty probes
    if(probeStruct->nPoints==0) return;

    //! if tAvg==1 the probe will be evaluated in every sub-timestep of each respective level
    //! else, the probe will only be evaluated in each synchronous time step tAvg

    uint level_coefficient = exp2(level);

    uint tAvg_level = this->tAvg==1 ? this->tAvg: this->tAvg*level_coefficient;
    uint tOut_level = this->tOut*level_coefficient;
    uint tStartOut_level = this->tStartOut*level_coefficient;
    uint tStartAvg_level = this->tStartAvg*level_coefficient;

    uint tAfterStartAvg = t_level - tStartAvg_level;
    uint tAfterStartOut = t_level - tStartOut_level;

    if( (t > this->tStartAvg) && (tAfterStartAvg % tAvg_level == 0))
    {
        this->calculateQuantities(probeStruct, para, t_level, level);

        if(t > this->tStartTmpAveraging) probeStruct->timestepInTimeAverage++;
        if(this->outputTimeSeries && (t_level >= tStartOut_level)) probeStruct->timestepInTimeseries++;
    }

    //! output only in synchronous timesteps
    if( (t > this->tStartOut) && (tAfterStartOut % tOut_level == 0) )
    {   
        if(this->hasDeviceQuantityArray)
            cudaMemoryManager->cudaCopyProbeQuantityArrayDtoH(this, level);
        this->write(para, level, t);
        
        if(level == 0&& !this->outputTimeSeries) this->writeParallelFile(para, t);

        if(this->outputTimeSeries)
        {
            probeStruct->lastTimestepInOldTimeseries = probeStruct->timestepInTimeseries > 0 ? probeStruct->timestepInTimeseries - 1: 0;
            probeStruct->timestepInTimeseries = 0;
        }
    }
}

void Probe::free(Parameter* para, CudaMemoryManager* cudaMemoryManager)
{
    for(int level=0; level<=para->getMaxLevel(); level++)
    {   
        if(this->probeParams[level]->hasDistances)
            cudaMemoryManager->cudaFreeProbeDistances(this, level);
        cudaMemoryManager->cudaFreeProbeIndices(this, level);
        cudaMemoryManager->cudaFreeProbeQuantityArray(this, level);
        cudaMemoryManager->cudaFreeProbeQuantitiesAndOffsets(this, level);
    }
}


void Probe::addStatistic(Statistic variable)
{
    if (!this->isAvailableStatistic(variable)) throw std::runtime_error("Probe::addStatistic(): Statistic not available for this probe type!");

    this->quantities[int(variable)] = true;
    switch(variable)
    {
        case Statistic::Variances: 
            this->addStatistic(Statistic::Means); break;

        default: break;
    }
}

template<typename T>
std::string nameComponent(std::string name, T value)
{
    return "_" + name + "_" + StringUtil::toString<T>(value); 
}

std::string Probe::makeParallelFileName(int id, int t)
{
    return this->probeName + "_bin" + nameComponent<int>("ID", id) + nameComponent<int>("t", t) + ".vtk"; 

}

std::string Probe::makeGridFileName(int level, int id, int t, uint part)
{
    return this->probeName + "_bin" + nameComponent<int>("lev", level)
                                    + nameComponent<int>("ID", id)
                                    + nameComponent<int>("part", part)
                                    + nameComponent<int>("t", t) + ".vtk";
}

std::string Probe::makeTimeseriesFileName(int level, int id)
{
    return this->probeName + "_timeseries" + nameComponent<int>("lev", level)
                                    + nameComponent<int>("ID", id)
                                    + ".txt";
}
void Probe::addAllAvailableStatistics()
{
    for( int var=0; var < int(Statistic::LAST); var++)
    {
        if(this->isAvailableStatistic(static_cast<Statistic>(var))) 
            this->addStatistic(static_cast<Statistic>(var));
    }
}

void Probe::write(Parameter* para, int level, int t)
{
    if(this->outputTimeSeries)
    {
        this->appendTimeseriesFile(para, level, t);
    }
    else
    {
        int t_write = this->fileNameLU ? t: t/this->tOut; 

        const uint numberOfParts = this->getProbeStruct(level)->nPoints / para->getlimitOfNodesForVTK() + 1;

        std::vector<std::string> fnames;
        for (uint i = 1; i <= numberOfParts; i++)
        {
            this->writeGridFile(para, level, t_write, i);
        }
    }

}

void Probe::writeParallelFile(Parameter* para, int t)
{
    int t_write = this->fileNameLU ? t: t/this->tOut; 
    std::string filename = this->outputPath + "/" + this->makeParallelFileName(para->getMyProcessID(), t_write);

    std::vector<std::string> nodedatanames = this->getVarNames();
    std::vector<std::string> cellNames;

    getWriter()->writeParallelFile(filename, fileNamesForCollectionFile, nodedatanames, cellNames);

    this->fileNamesForCollectionFile.clear();
}

void Probe::writeGridFile(Parameter* para, int level, int t, uint part)
{
    std::string fname = this->outputPath + "/" + this->makeGridFileName(level, para->getMyProcessID(), t, part);

    std::vector< UbTupleFloat3 > nodes;
    std::vector< std::string > nodedatanames = this->getVarNames();

    std::vector< std::vector< double > > nodedata(nodedatanames.size());

    SPtr<ProbeStruct> probeStruct = this->getProbeStruct(level);

    uint startpos = (part-1) * para->getlimitOfNodesForVTK();
    uint sizeOfNodes = min(para->getlimitOfNodesForVTK(), probeStruct->nPoints - startpos);
    uint endpos = startpos + sizeOfNodes;

    //////////////////////////////////////////////////////////////////////////
    nodes.resize(sizeOfNodes);

    for (uint pos = startpos; pos < endpos; pos++)
    {
        nodes[pos-startpos] = makeUbTuple(  float(probeStruct->pointCoordsX[pos]),
                                            float(probeStruct->pointCoordsY[pos]),
                                            float(probeStruct->pointCoordsZ[pos]));
    }

    for( auto it=nodedata.begin(); it!=nodedata.end(); it++) it->resize(sizeOfNodes);

    uint arrLen = probeStruct->nPoints;
    int nTimesteps = probeStruct->nTimesteps;
    int timestep = probeStruct->timestepInTimeseries;

    for( int var=0; var < int(Statistic::LAST); var++)
    {           
        if(this->quantities[var])
        {

            Statistic statistic = static_cast<Statistic>(var);
            std::vector<PostProcessingVariable> postProcessingVariables = this->getPostProcessingVariables(statistic);
            uint n_arrs = uint(postProcessingVariables.size());

            uint arrOff = probeStruct->arrayOffsetsH[var];

            for(uint arr=0; arr<n_arrs; arr++)
            {
                real coeff = postProcessingVariables[arr].conversionFactor(level);
                
                for (uint pos = startpos; pos < endpos; pos++)
                {
                    nodedata[arrOff+arr][pos-startpos] = double(probeStruct->quantitiesArrayH[calcArrayIndex(pos, arrLen, timestep, nTimesteps, arrOff+arr)]*coeff);
                }
            }
        }
    }
    std::string fullName = getWriter()->writeNodesWithNodeData(fname, nodes, nodedatanames, nodedata);
    this->fileNamesForCollectionFile.push_back(fullName.substr(fullName.find_last_of('/') + 1));
}

std::string Probe::writeTimeseriesHeader(Parameter* para, int level)
{
/*
File Layout:
TimeseriesOutput
Quantities: Quant1 Quant2 Quant3
Positions:
point1.x, point1.y, point1.z
point2.x, point2.y, point2.z
...
t0 point1.quant1 point2.quant1 ... point1.quant2 point2.quant2 ...
t1 point1.quant1 point2.quant1 ... point1.quant2 point2.quant2 ...
*/
    auto probeStruct = this->getProbeStruct(level);
    std::string fname = this->outputPath + "/" + this->makeTimeseriesFileName(level, para->getMyProcessID());
    std::ofstream out(fname.c_str(), std::ios::out | std::ios::binary);

    if(!out.is_open()) throw std::runtime_error("Could not open timeseries file!");

    out << "TimeseriesOutput \n";
    out << "Quantities: ";
    for(std::string name : getVarNames())
        out << name << ", ";
    out << "\n";
    out << "Number of points in this file: \n";
    out << probeStruct->nPoints << "\n";
    out << "Positions: x, y, z\n";
    for( uint i=0; i<probeStruct->nPoints; i++)
        out << probeStruct->pointCoordsX[i] << ", " << probeStruct->pointCoordsY[i] << ", " << probeStruct->pointCoordsZ[i] << "\n";

    out.close();

    return fname;
}

void Probe::appendTimeseriesFile(Parameter* para, int level, int t)
{
    std::ofstream out(this->timeseriesFileNames[level], std::ios::app | std::ios::binary);

    uint t_level = para->getTimeStep(level, t, false);
    uint tAvg_level = this->tAvg==1 ? this->tAvg: this->tAvg*exp2(-level);

    real dt = para->getTimeRatio()*tAvg_level;
    auto probeStruct = this->getProbeStruct(level);

    real t_start = ( t-this->tOut )*para->getTimeRatio();

    int vals_per_timestep = probeStruct->nPoints*probeStruct->nArrays+1;

    real* timestep_array = (real*) malloc(vals_per_timestep*sizeof(real));

    for(uint timestep=0; timestep<probeStruct->timestepInTimeseries; timestep++)
    {
        int val = 0;
        timestep_array[val] = t_start+timestep*dt;
        val++;

        for( int var=0; var < int(Statistic::LAST); var++)
        {           
            if(!this->quantities[var]) continue;
            
            Statistic statistic = static_cast<Statistic>(var);
            std::vector<PostProcessingVariable> postProcessingVariables = this->getPostProcessingVariables(statistic);
            uint n_arrs = uint(postProcessingVariables.size());

            uint arrOff = probeStruct->arrayOffsetsH[var];

            for(uint arr=0; arr<n_arrs; arr++)
            {
                real coeff = postProcessingVariables[arr].conversionFactor(level);
                for(uint point=0; point<probeStruct->nPoints; point++)
                {
                    timestep_array[val] = probeStruct->quantitiesArrayH[calcArrayIndex(point, probeStruct->nPoints, timestep, probeStruct->nTimesteps, arrOff+arr)]*coeff;
                    val++;
                }
            }
            
        }
        out.write((char*) timestep_array, sizeof(real)*vals_per_timestep);
    }
    out.close();
}



std::vector<std::string> Probe::getVarNames()
{
    std::vector<std::string> varNames;
    for( int statistic=0; statistic < int(Statistic::LAST); statistic++)
    {
        if(this->quantities[statistic])
        {
            std::vector<PostProcessingVariable> postProcessingVariables = this->getPostProcessingVariables(static_cast<Statistic>(statistic));            
            for(int i = 0; i<postProcessingVariables.size(); i++) 
                varNames.push_back(postProcessingVariables[i].name);
        }
    }
    return varNames;
}