#include "hip/hip_runtime.h"
#include "Probe.h"
#include "PointProbe.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cuda/CudaGrid.h>

#include "Parameter/Parameter.h"
#include "DataStructureInitializer/GridProvider.h"
#include "GPU/CudaMemoryManager.h"

bool PointProbe::isAvailableStatistic(Statistic _variable)
{
    bool isAvailable;
    switch (_variable)
    {
        case Statistic::Instantaneous:
        case Statistic::Means:
        case Statistic::Variances:
            isAvailable = true;
            break;
        case Statistic::SpatialMeans:
        case Statistic::SpatioTemporalMeans:
        case Statistic::SpatialCovariances:
        case Statistic::SpatioTemporalCovariances:
        case Statistic::SpatialSkewness:
        case Statistic::SpatioTemporalSkewness:
        case Statistic::SpatialFlatness:
        case Statistic::SpatioTemporalFlatness:
            isAvailable = false;
            break;
        default:
            isAvailable = false;
    }
    return isAvailable;
}

std::vector<PostProcessingVariable> PointProbe::getPostProcessingVariables(Statistic statistic)
{
    std::vector<PostProcessingVariable> postProcessingVariables;
    switch (statistic)
    {
    case Statistic::Instantaneous:
        postProcessingVariables.push_back( PostProcessingVariable("vx",  velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho", this->densityRatio ) );
        break;
    case Statistic::Means:
        postProcessingVariables.push_back( PostProcessingVariable("vx_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho_mean", this->densityRatio ) );
        break;
    case Statistic::Variances:
        postProcessingVariables.push_back( PostProcessingVariable("vx_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho_var", this->densityRatio) );
        break;

    default:
        throw std::runtime_error("PointProbe::getPostProcessingVariables: Statistic unavailable!");
        break;
    }
    return postProcessingVariables;
}

void PointProbe::findPoints(Parameter* para, GridProvider* gridProvider, std::vector<int>& probeIndices_level,
                       std::vector<real>& distX_level, std::vector<real>& distY_level, std::vector<real>& distZ_level,      
                       std::vector<real>& pointCoordsX_level, std::vector<real>& pointCoordsY_level, std::vector<real>& pointCoordsZ_level,
                       int level)
{

    real dx = abs(para->getParH(level)->coordinateX[1]-para->getParH(level)->coordinateX[para->getParH(level)->neighborX[1]]);
    for(uint j=1; j<para->getParH(level)->numberOfNodes; j++ )
    {    
        for(uint point=0; point<this->pointCoordsX.size(); point++)
        {
            real pointCoordX = this->pointCoordsX[point];
            real pointCoordY = this->pointCoordsY[point];
            real pointCoordZ = this->pointCoordsZ[point];
            real distX = pointCoordX-para->getParH(level)->coordinateX[j];
            real distY = pointCoordY-para->getParH(level)->coordinateY[j];
            real distZ = pointCoordZ-para->getParH(level)->coordinateZ[j];
            if( distX <=dx && distY <=dx && distZ <=dx &&
                distX >0.f && distY >0.f && distZ >0.f)
            {
                probeIndices_level.push_back(j);
                distX_level.push_back( distX/dx );
                distY_level.push_back( distY/dx );
                distZ_level.push_back( distZ/dx );
                pointCoordsX_level.push_back( pointCoordX );
                pointCoordsY_level.push_back( pointCoordY );
                pointCoordsZ_level.push_back( pointCoordZ );
            }
        }
    }
}

void PointProbe::calculateQuantities(SPtr<ProbeStruct> probeStruct, Parameter* para, uint t, int level)
{
    vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(para->getParH(level)->numberofthreads, probeStruct->nPoints);
    interpAndCalcQuantitiesKernel<<<grid.grid, grid.threads>>>(  probeStruct->pointIndicesD, probeStruct->nPoints, probeStruct->vals,
                                                probeStruct->distXD, probeStruct->distYD, probeStruct->distZD,
                                                para->getParD(level)->velocityX, para->getParD(level)->velocityY, para->getParD(level)->velocityZ, para->getParD(level)->rho, 
                                                para->getParD(level)->neighborX, para->getParD(level)->neighborY, para->getParD(level)->neighborZ, 
                                                probeStruct->quantitiesD, probeStruct->arrayOffsetsD, probeStruct->quantitiesArrayD);
}

void PointProbe::addProbePointsFromList(std::vector<real>& _pointCoordsX, std::vector<real>& _pointCoordsY, std::vector<real>& _pointCoordsZ)
{
    bool isSameLength = ( (_pointCoordsX.size()==_pointCoordsY.size()) && (_pointCoordsY.size()==_pointCoordsZ.size()));
    if (!isSameLength) throw std::runtime_error("Probe::addProbePointsFromList(): point lists have different lengths!");
    this->pointCoordsX.insert(this->pointCoordsX.end(), _pointCoordsX.begin(),  _pointCoordsX.end());
    this->pointCoordsY.insert(this->pointCoordsY.end(), _pointCoordsY.begin(),  _pointCoordsY.end());
    this->pointCoordsZ.insert(this->pointCoordsZ.end(), _pointCoordsZ.begin(),  _pointCoordsZ.end());
    printf("Added list of %u  points \n", uint(_pointCoordsX.size()) );
}

void PointProbe::addProbePointsFromXNormalPlane(real pos_x, real pos0_y, real pos0_z, real pos1_y, real pos1_z, uint n_y, uint n_z)
{
    int delta_y = (pos1_y-pos0_y)/(n_y-1);
    int delta_z = (pos1_z-pos0_z)/(n_z-1);

    this->pointCoordsX.reserve(this->pointCoordsX.size()+n_y*n_z);
    this->pointCoordsY.reserve(this->pointCoordsY.size()+n_y*n_z);
    this->pointCoordsZ.reserve(this->pointCoordsZ.size()+n_y*n_z);

    for(int n_y=0; n_y<n_y; n_y++)
    {
        for(int n_z=0; n_z<n_z; n_z++)
        {
            this->pointCoordsX.push_back(pos_x);
            this->pointCoordsY.push_back(pos0_y+delta_y*n_y);
            this->pointCoordsZ.push_back(pos0_z+delta_z*n_z);
        }
    }
    printf("Added %u  points \n",  n_y*n_z);

}

void PointProbe::getTaggedFluidNodes(Parameter *para, GridProvider* gridProvider)
{
    for(int level=0; level<=para->getMaxLevel(); level++)
    {
        SPtr<ProbeStruct> probeStruct = this->getProbeStruct(level);
        std::vector<uint> probeIndices( probeStruct->pointIndicesH, probeStruct->pointIndicesH+probeStruct->nIndices);
        gridProvider->tagFluidNodeIndices( probeIndices, CollisionTemplate::WriteMacroVars, level);
    }
}