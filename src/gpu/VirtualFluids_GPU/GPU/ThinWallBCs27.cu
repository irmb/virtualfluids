#include "hip/hip_runtime.h"
//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ / 
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////

/* Device code */
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;

/////////////////////////////////////////////////////////////////////////
__global__ void QVelDeviceCompThinWallsPartOne27(
	real* vx,
	real* vy,
	real* vz,
	real* DD, 
	int* k_Q, 
	real* QQ,
	uint numberOfBCnodes, 
	real om1, 
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	unsigned long long numberOfLBnodes, 
	bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dP00] = &DD[dP00 * numberOfLBnodes];
      D.f[dM00] = &DD[dM00 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
   } 
   else
   {
      D.f[dM00] = &DD[dP00 * numberOfLBnodes];
      D.f[dP00] = &DD[dM00 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real VeloX = vx[k];
      real VeloY = vy[k];
      real VeloZ = vz[k]; 
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dP00 * numberOfBCnodes];
      q_dirW   = &QQ[dM00 * numberOfBCnodes];
      q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      uint KQK  = k_Q[k];
      uint kzero= KQK;
      uint ke   = KQK;
      uint kw   = neighborX[KQK];
      uint kn   = KQK;
      uint ks   = neighborY[KQK];
      uint kt   = KQK;
      uint kb   = neighborZ[KQK];
      uint ksw  = neighborY[kw];
      uint kne  = KQK;
      uint kse  = ks;
      uint knw  = kw;
      uint kbw  = neighborZ[kw];
      uint kte  = KQK;
      uint kbe  = kb;
      uint ktw  = kw;
      uint kbs  = neighborZ[ks];
      uint ktn  = KQK;
      uint kbn  = kb;
      uint kts  = ks;
      uint ktse = ks;
      uint kbnw = kbw;
      uint ktnw = kw;
      uint kbse = kbs;
      uint ktsw = ksw;
      uint kbne = kb;
      uint ktne = KQK;
      uint kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dP00])[ke   ];
      f_E    = (D.f[dM00])[kw   ];
      f_S    = (D.f[DIR_0P0])[kn   ];
      f_N    = (D.f[DIR_0M0])[ks   ];
      f_B    = (D.f[DIR_00P])[kt   ];
      f_T    = (D.f[DIR_00M])[kb   ];
      f_SW   = (D.f[DIR_PP0])[kne  ];
      f_NE   = (D.f[DIR_MM0])[ksw  ];
      f_NW   = (D.f[DIR_PM0])[kse  ];
      f_SE   = (D.f[DIR_MP0])[knw  ];
      f_BW   = (D.f[DIR_P0P])[kte  ];
      f_TE   = (D.f[DIR_M0M])[kbw  ];
      f_TW   = (D.f[DIR_P0M])[kbe  ];
      f_BE   = (D.f[DIR_M0P])[ktw  ];
      f_BS   = (D.f[DIR_0PP])[ktn  ];
      f_TN   = (D.f[DIR_0MM])[kbs  ];
      f_TS   = (D.f[DIR_0PM])[kbn  ];
      f_BN   = (D.f[DIR_0MP])[kts  ];
      f_BSW  = (D.f[DIR_PPP])[ktne ];
      f_BNE  = (D.f[DIR_MMP])[ktsw ];
      f_BNW  = (D.f[DIR_PMP])[ktse ];
      f_BSE  = (D.f[DIR_MPP])[ktnw ];
      f_TSW  = (D.f[DIR_PPM])[kbne ];
      f_TNE  = (D.f[DIR_MMM])[kbsw ];
      f_TNW  = (D.f[DIR_PMM])[kbse ];
      f_TSE  = (D.f[DIR_MPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho + c9o2 * ( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq);
		 (D.f[dM00])[kw] = (c1o1 - q) / (c1o1 + q)*(f_E - f_W + (f_E + f_W - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_E + f_W) - c6o1*c2o27*(VeloX)) / (c1o1 + q);
	  }

	  q = q_dirW[k];
	  if (q >= c0o1 && q <= c1o1)
	  {
		  feq = c2o27* (drho + c9o2 * (-vx1)*(-vx1) * (c1o1 + drho) - cu_sq);
		  (D.f[dP00])[ke] = (c1o1 - q) / (c1o1 + q)*(f_W - f_E + (f_W + f_E - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_W + f_E) - c6o1*c2o27*(-VeloX)) / (c1o1 + q);
	  }

	  q = q_dirN[k];
	  if (q >= c0o1 && q <= c1o1)
	  {
		  feq = c2o27* (drho + c9o2 * (vx2)*(vx2) * (c1o1 + drho) - cu_sq);
		  (D.f[DIR_0M0])[ks] = (c1o1 - q) / (c1o1 + q)*(f_N - f_S + (f_N + f_S - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_N + f_S) - c6o1*c2o27*(VeloY)) / (c1o1 + q);
	  }

	  q = q_dirS[k];
	  if (q >= c0o1 && q <= c1o1)
	  {
		  feq = c2o27* (drho + c9o2 * (-vx2)*(-vx2) * (c1o1 + drho) - cu_sq);
		  (D.f[DIR_0P0])[kn] = (c1o1 - q) / (c1o1 + q)*(f_S - f_N + (f_S + f_N - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_S + f_N) - c6o1*c2o27*(-VeloY)) / (c1o1 + q);
	  }

	  q = q_dirT[k];
	  if (q >= c0o1 && q <= c1o1)
	  {
		  feq = c2o27* (drho + c9o2 * (vx3)*(vx3) * (c1o1 + drho) - cu_sq);
		  (D.f[DIR_00M])[kb] = (c1o1 - q) / (c1o1 + q)*(f_T - f_B + (f_T + f_B - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_T + f_B) - c6o1*c2o27*(VeloZ)) / (c1o1 + q);
	  }

	  q = q_dirB[k];
	  if (q >= c0o1 && q <= c1o1)
	  {
		  feq = c2o27* (drho + c9o2 * (-vx3)*(-vx3) * (c1o1 + drho) - cu_sq);
		  (D.f[DIR_00P])[kt] = (c1o1 - q) / (c1o1 + q)*(f_B - f_T + (f_B + f_T - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_B + f_T) - c6o1*c2o27*(-VeloZ)) / (c1o1 + q);
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * ( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_MM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*( VeloX+VeloY))/(c1o1+q);
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * (-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_PP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * ( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_MP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * (-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_PM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * ( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_M0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * (-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_P0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * ( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_M0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * (-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_P0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * (     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * (    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*(-VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * (     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho + c9o2 * (    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq);
         (D.f[DIR_0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*(-VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho + c9o2 * ( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho + c9o2 * (-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho + c9o2 * ( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho + c9o2 * (-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho + c9o2 * ( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho + c9o2 * (-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho + c9o2 * ( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho + c9o2 * (-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
__global__ void QDeviceCompThinWallsPartOne27(
	real* DD,
	int* k_Q,
	real* QQ,
	unsigned int numberOfBCnodes,
	real om1,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned long long numberOfLBnodes,
	bool isEvenTimestep)
{
	Distributions27 D;
	if (isEvenTimestep == true)
	{
		D.f[dP00] = &DD[dP00 * numberOfLBnodes];
		D.f[dM00] = &DD[dM00 * numberOfLBnodes];
		D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
		D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
		D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
		D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
		D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
		D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
		D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
		D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
		D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
		D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
		D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
		D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
		D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
		D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
		D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
		D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
		D.f[d000] = &DD[d000 * numberOfLBnodes];
		D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
		D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
		D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
		D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
		D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
		D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
		D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
		D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
	}
	else
	{
		D.f[dM00] = &DD[dP00 * numberOfLBnodes];
		D.f[dP00] = &DD[dM00 * numberOfLBnodes];
		D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
		D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
		D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
		D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
		D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
		D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
		D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
		D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
		D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
		D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
		D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
		D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
		D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
		D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
		D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
		D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
		D.f[d000] = &DD[d000 * numberOfLBnodes];
		D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
		D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
		D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
		D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
		D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
		D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
		D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
		D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
	}
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k < numberOfBCnodes)
	{
		////////////////////////////////////////////////////////////////////////////////
		real *q_dirE, *q_dirW, *q_dirN, *q_dirS, *q_dirT, *q_dirB,
			*q_dirNE, *q_dirSW, *q_dirSE, *q_dirNW, *q_dirTE, *q_dirBW,
			*q_dirBE, *q_dirTW, *q_dirTN, *q_dirBS, *q_dirBN, *q_dirTS,
			*q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
			*q_dirBSE, *q_dirBNW;
		q_dirE = &QQ[dP00 * numberOfBCnodes];
		q_dirW = &QQ[dM00 * numberOfBCnodes];
		q_dirN = &QQ[DIR_0P0 * numberOfBCnodes];
		q_dirS = &QQ[DIR_0M0 * numberOfBCnodes];
		q_dirT = &QQ[DIR_00P * numberOfBCnodes];
		q_dirB = &QQ[DIR_00M * numberOfBCnodes];
		q_dirNE = &QQ[DIR_PP0 * numberOfBCnodes];
		q_dirSW = &QQ[DIR_MM0 * numberOfBCnodes];
		q_dirSE = &QQ[DIR_PM0 * numberOfBCnodes];
		q_dirNW = &QQ[DIR_MP0 * numberOfBCnodes];
		q_dirTE = &QQ[DIR_P0P * numberOfBCnodes];
		q_dirBW = &QQ[DIR_M0M * numberOfBCnodes];
		q_dirBE = &QQ[DIR_P0M * numberOfBCnodes];
		q_dirTW = &QQ[DIR_M0P * numberOfBCnodes];
		q_dirTN = &QQ[DIR_0PP * numberOfBCnodes];
		q_dirBS = &QQ[DIR_0MM * numberOfBCnodes];
		q_dirBN = &QQ[DIR_0PM * numberOfBCnodes];
		q_dirTS = &QQ[DIR_0MP * numberOfBCnodes];
		q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
		q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
		q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
		q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
		q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
		q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
		q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
		q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
		////////////////////////////////////////////////////////////////////////////////
		//index
		unsigned int KQK = k_Q[k];
		unsigned int kzero = KQK;
		unsigned int ke = KQK;
		unsigned int kw = neighborX[KQK];
		unsigned int kn = KQK;
		unsigned int ks = neighborY[KQK];
		unsigned int kt = KQK;
		unsigned int kb = neighborZ[KQK];
		unsigned int ksw = neighborY[kw];
		unsigned int kne = KQK;
		unsigned int kse = ks;
		unsigned int knw = kw;
		unsigned int kbw = neighborZ[kw];
		unsigned int kte = KQK;
		unsigned int kbe = kb;
		unsigned int ktw = kw;
		unsigned int kbs = neighborZ[ks];
		unsigned int ktn = KQK;
		unsigned int kbn = kb;
		unsigned int kts = ks;
		unsigned int ktse = ks;
		unsigned int kbnw = kbw;
		unsigned int ktnw = kw;
		unsigned int kbse = kbs;
		unsigned int ktsw = ksw;
		unsigned int kbne = kb;
		unsigned int ktne = KQK;
		unsigned int kbsw = neighborZ[ksw];
		////////////////////////////////////////////////////////////////////////////////
		real f_E, f_W, f_N, f_S, f_T, f_B, f_NE, f_SW, f_SE, f_NW, f_TE, f_BW, f_BE,
			f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

		f_W = (D.f[dP00])[ke];
		f_E = (D.f[dM00])[kw];
		f_S = (D.f[DIR_0P0])[kn];
		f_N = (D.f[DIR_0M0])[ks];
		f_B = (D.f[DIR_00P])[kt];
		f_T = (D.f[DIR_00M])[kb];
		f_SW = (D.f[DIR_PP0])[kne];
		f_NE = (D.f[DIR_MM0])[ksw];
		f_NW = (D.f[DIR_PM0])[kse];
		f_SE = (D.f[DIR_MP0])[knw];
		f_BW = (D.f[DIR_P0P])[kte];
		f_TE = (D.f[DIR_M0M])[kbw];
		f_TW = (D.f[DIR_P0M])[kbe];
		f_BE = (D.f[DIR_M0P])[ktw];
		f_BS = (D.f[DIR_0PP])[ktn];
		f_TN = (D.f[DIR_0MM])[kbs];
		f_TS = (D.f[DIR_0PM])[kbn];
		f_BN = (D.f[DIR_0MP])[kts];
		f_BSW = (D.f[DIR_PPP])[ktne];
		f_BNE = (D.f[DIR_MMP])[ktsw];
		f_BNW = (D.f[DIR_PMP])[ktse];
		f_BSE = (D.f[DIR_MPP])[ktnw];
		f_TSW = (D.f[DIR_PPM])[kbne];
		f_TNE = (D.f[DIR_MMM])[kbsw];
		f_TNW = (D.f[DIR_PMM])[kbse];
		f_TSE = (D.f[DIR_MPM])[kbnw];
		////////////////////////////////////////////////////////////////////////////////
		real vx1, vx2, vx3, drho, feq, q;
		drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
			f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW +
			f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]);

		vx1 = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
			((f_BE - f_TW) + (f_TE - f_BW)) + ((f_SE - f_NW) + (f_NE - f_SW)) +
			(f_E - f_W)) / (c1o1 + drho);


		vx2 = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
			((f_BN - f_TS) + (f_TN - f_BS)) + (-(f_SE - f_NW) + (f_NE - f_SW)) +
			(f_N - f_S)) / (c1o1 + drho);

		vx3 = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
			(-(f_BN - f_TS) + (f_TN - f_BS)) + ((f_TE - f_BW) - (f_BE - f_TW)) +
			(f_T - f_B)) / (c1o1 + drho);

		////////////////////////////////////////////////////////////////////////////////
		real cu_sq = c3o2*(vx1*vx1 + vx2*vx2 + vx3*vx3) * (c1o1 + drho);
		////////////////////////////////////////////////////////////////////////////////

		q = q_dirE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c2o27* (drho + c9o2*(vx1)*(vx1) * (c1o1 + drho) - cu_sq);
			(D.f[dM00])[kw] = (c1o1 - q) / (c1o1 + q)*(f_E - f_W + (f_E + f_W - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_E + f_W)) / (c1o1 + q);
		}

		q = q_dirW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c2o27* (drho + c9o2*(-vx1)*(-vx1) * (c1o1 + drho) - cu_sq);
			(D.f[dP00])[ke] = (c1o1 - q) / (c1o1 + q)*(f_W - f_E + (f_W + f_E - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_W + f_E)) / (c1o1 + q);
		}

		q = q_dirN[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c2o27* (drho + c9o2*(vx2)*(vx2) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_0M0])[ks] = (c1o1 - q) / (c1o1 + q)*(f_N - f_S + (f_N + f_S - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_N + f_S)) / (c1o1 + q);
		}

		q = q_dirS[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c2o27* (drho + c9o2*(-vx2)*(-vx2) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_0P0])[kn] = (c1o1 - q) / (c1o1 + q)*(f_S - f_N + (f_S + f_N - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_S + f_N)) / (c1o1 + q);
		}

		q = q_dirT[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c2o27* (drho + c9o2*(vx3)*(vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_00M])[kb] = (c1o1 - q) / (c1o1 + q)*(f_T - f_B + (f_T + f_B - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_T + f_B)) / (c1o1 + q);
		}

		q = q_dirB[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c2o27* (drho + c9o2*(-vx3)*(-vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_00P])[kt] = (c1o1 - q) / (c1o1 + q)*(f_B - f_T + (f_B + f_T - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_B + f_T)) / (c1o1 + q);
		}

		q = q_dirNE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(vx1 + vx2)*(vx1 + vx2) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_MM0])[ksw] = (c1o1 - q) / (c1o1 + q)*(f_NE - f_SW + (f_NE + f_SW - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_NE + f_SW)) / (c1o1 + q);
		}

		q = q_dirSW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(-vx1 - vx2)*(-vx1 - vx2) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_PP0])[kne] = (c1o1 - q) / (c1o1 + q)*(f_SW - f_NE + (f_SW + f_NE - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_SW + f_NE)) / (c1o1 + q);
		}

		q = q_dirSE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(vx1 - vx2)*(vx1 - vx2) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_MP0])[knw] = (c1o1 - q) / (c1o1 + q)*(f_SE - f_NW + (f_SE + f_NW - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_SE + f_NW)) / (c1o1 + q);
		}

		q = q_dirNW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(-vx1 + vx2)*(-vx1 + vx2) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_PM0])[kse] = (c1o1 - q) / (c1o1 + q)*(f_NW - f_SE + (f_NW + f_SE - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_NW + f_SE)) / (c1o1 + q);
		}

		q = q_dirTE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(vx1 + vx3)*(vx1 + vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_M0M])[kbw] = (c1o1 - q) / (c1o1 + q)*(f_TE - f_BW + (f_TE + f_BW - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_TE + f_BW)) / (c1o1 + q);
		}

		q = q_dirBW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(-vx1 - vx3)*(-vx1 - vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_P0P])[kte] = (c1o1 - q) / (c1o1 + q)*(f_BW - f_TE + (f_BW + f_TE - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_BW + f_TE)) / (c1o1 + q);
		}

		q = q_dirBE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(vx1 - vx3)*(vx1 - vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_M0P])[ktw] = (c1o1 - q) / (c1o1 + q)*(f_BE - f_TW + (f_BE + f_TW - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_BE + f_TW)) / (c1o1 + q);
		}

		q = q_dirTW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(-vx1 + vx3)*(-vx1 + vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_P0M])[kbe] = (c1o1 - q) / (c1o1 + q)*(f_TW - f_BE + (f_TW + f_BE - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_TW + f_BE)) / (c1o1 + q);
		}

		q = q_dirTN[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(vx2 + vx3)*(vx2 + vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_0MM])[kbs] = (c1o1 - q) / (c1o1 + q)*(f_TN - f_BS + (f_TN + f_BS - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_TN + f_BS)) / (c1o1 + q);
		}

		q = q_dirBS[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(-vx2 - vx3)*(-vx2 - vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_0PP])[ktn] = (c1o1 - q) / (c1o1 + q)*(f_BS - f_TN + (f_BS + f_TN - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_BS + f_TN)) / (c1o1 + q);
		}

		q = q_dirBN[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(vx2 - vx3)*(vx2 - vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_0MP])[kts] = (c1o1 - q) / (c1o1 + q)*(f_BN - f_TS + (f_BN + f_TS - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_BN + f_TS)) / (c1o1 + q);
		}

		q = q_dirTS[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o54* (drho + c9o2*(-vx2 + vx3)*(-vx2 + vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_0PM])[kbn] = (c1o1 - q) / (c1o1 + q)*(f_TS - f_BN + (f_TS + f_BN - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_TS + f_BN)) / (c1o1 + q);
		}

		q = q_dirTNE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o216*(drho + c9o2*(vx1 + vx2 + vx3)*(vx1 + vx2 + vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_MMM])[kbsw] = (c1o1 - q) / (c1o1 + q)*(f_TNE - f_BSW + (f_TNE + f_BSW - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_TNE + f_BSW)) / (c1o1 + q);
		}

		q = q_dirBSW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o216*(drho + c9o2*(-vx1 - vx2 - vx3)*(-vx1 - vx2 - vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_PPP])[ktne] = (c1o1 - q) / (c1o1 + q)*(f_BSW - f_TNE + (f_BSW + f_TNE - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_BSW + f_TNE)) / (c1o1 + q);
		}

		q = q_dirBNE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o216*(drho + c9o2*(vx1 + vx2 - vx3)*(vx1 + vx2 - vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_MMP])[ktsw] = (c1o1 - q) / (c1o1 + q)*(f_BNE - f_TSW + (f_BNE + f_TSW - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_BNE + f_TSW)) / (c1o1 + q);
		}

		q = q_dirTSW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o216*(drho + c9o2*(-vx1 - vx2 + vx3)*(-vx1 - vx2 + vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_PPM])[kbne] = (c1o1 - q) / (c1o1 + q)*(f_TSW - f_BNE + (f_TSW + f_BNE - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_TSW + f_BNE)) / (c1o1 + q);
		}

		q = q_dirTSE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o216*(drho + c9o2*(vx1 - vx2 + vx3)*(vx1 - vx2 + vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_MPM])[kbnw] = (c1o1 - q) / (c1o1 + q)*(f_TSE - f_BNW + (f_TSE + f_BNW - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_TSE + f_BNW)) / (c1o1 + q);
		}

		q = q_dirBNW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o216*(drho + c9o2*(-vx1 + vx2 - vx3)*(-vx1 + vx2 - vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_PMP])[ktse] = (c1o1 - q) / (c1o1 + q)*(f_BNW - f_TSE + (f_BNW + f_TSE - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_BNW + f_TSE)) / (c1o1 + q);
		}

		q = q_dirBSE[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o216*(drho + c9o2*(vx1 - vx2 - vx3)*(vx1 - vx2 - vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_MPP])[ktnw] = (c1o1 - q) / (c1o1 + q)*(f_BSE - f_TNW + (f_BSE + f_TNW - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_BSE + f_TNW)) / (c1o1 + q);
		}

		q = q_dirTNW[k];
		if (q >= c0o1 && q <= c1o1)
		{
			feq = c1o216*(drho + c9o2*(-vx1 + vx2 + vx3)*(-vx1 + vx2 + vx3) * (c1o1 + drho) - cu_sq);
			(D.f[DIR_PMM])[kbse] = (c1o1 - q) / (c1o1 + q)*(f_TNW - f_BSE + (f_TNW + f_BSE - c2o1*feq*om1) / (c1o1 - om1))*c1o2 + (q*(f_TNW + f_BSE)) / (c1o1 + q);
		}
	}
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
__global__ void QThinWallsPartTwo27(
	real* DD, 
	int* k_Q, 
	real* QQ,
	uint numberOfBCnodes, 
	uint* geom,
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	uint* neighborWSB,
	unsigned long long numberOfLBnodes, 
	bool isEvenTimestep)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dP00 * numberOfBCnodes];
      q_dirW   = &QQ[dM00 * numberOfBCnodes];
      q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      uint KQK  = k_Q[k];
      //uint kzero= KQK;
      uint ke   = KQK;
      uint kw   = neighborX[KQK];
      uint kn   = KQK;
      uint ks   = neighborY[KQK];
      uint kt   = KQK;
      uint kb   = neighborZ[KQK];
      uint ksw  = neighborY[kw];
      uint kne  = KQK;
      uint kse  = ks;
      uint knw  = kw;
      uint kbw  = neighborZ[kw];
      uint kte  = KQK;
      uint kbe  = kb;
      uint ktw  = kw;
      uint kbs  = neighborZ[ks];
      uint ktn  = KQK;
      uint kbn  = kb;
      uint kts  = ks;
      uint ktse = ks;
      uint kbnw = kbw;
      uint ktnw = kw;
      uint kbse = kbs;
      uint ktsw = ksw;
      uint kbne = kb;
      uint ktne = KQK;
      uint kbsw = neighborZ[ksw];
	  ////////////////////////////////////////////////////////////////////////////////
	  //anti ET intermediate steps
	  uint kmmm = neighborWSB[KQK]; // -1 -1 -1
	  uint k0mm = neighborX[kmmm];  //  0 -1 -1
	  uint km0m = neighborY[kmmm];  // -1  0 -1
	  uint kmm0 = neighborZ[kmmm];  // -1 -1  0
	  uint k0m0 = neighborX[kmm0];  //  0 -1  0
	  uint km00 = neighborY[kmm0];  // -1  0  0
	  /////////////////////////////////////////////////
	  //final indices for anti ET
	  uint kpmm = neighborX[k0mm];  //  1 -1 -1
	  uint kmpm = neighborY[km0m];  // -1  1 -1
	  uint kmmp = neighborZ[kmm0];  // -1 -1  1
	  uint kmp0 = neighborY[km00];  // -1  1  0
	  uint km0p = neighborZ[km00];  // -1  0  1
	  uint k0mp = neighborZ[k0m0];  //  0 -1  1
	  ////////////////////////////////////////////////////////////////////////////////
	  Distributions27 D, DN;
	  if (isEvenTimestep == true)
	  {
		  D.f[dP00] = &DD[dP00 * numberOfLBnodes];
		  D.f[dM00] = &DD[dM00 * numberOfLBnodes];
		  D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
		  D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
		  D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
		  D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
		  D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
		  D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
		  D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
		  D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
		  D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
		  D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
		  D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
		  D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
		  D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
		  D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
		  D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
		  D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
		  D.f[d000] = &DD[d000 * numberOfLBnodes];
		  D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
		  D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
		  D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
		  D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
		  D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
		  D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
		  D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
		  D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
	  }
	  else
	  {
		  D.f[dM00] = &DD[dP00 * numberOfLBnodes];
		  D.f[dP00] = &DD[dM00 * numberOfLBnodes];
		  D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
		  D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
		  D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
		  D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
		  D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
		  D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
		  D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
		  D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
		  D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
		  D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
		  D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
		  D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
		  D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
		  D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
		  D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
		  D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
		  D.f[d000] = &DD[d000 * numberOfLBnodes];
		  D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
		  D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
		  D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
		  D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
		  D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
		  D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
		  D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
		  D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
	  }
	  if (isEvenTimestep==false)
      {
         DN.f[dP00] = &DD[dP00 * numberOfLBnodes];
         DN.f[dM00] = &DD[dM00 * numberOfLBnodes];
         DN.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
         DN.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
         DN.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
         DN.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
         DN.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
         DN.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
         DN.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
         DN.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
         DN.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
         DN.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
         DN.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
         DN.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
         DN.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
         DN.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
         DN.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
         DN.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
         DN.f[d000] = &DD[d000 * numberOfLBnodes];
         DN.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
         DN.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
         DN.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
         DN.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
         DN.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
         DN.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
         DN.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
         DN.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
      } 
      else
      {
         DN.f[dM00] = &DD[dP00 * numberOfLBnodes];
         DN.f[dP00] = &DD[dM00 * numberOfLBnodes];
         DN.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
         DN.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
         DN.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
         DN.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
         DN.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
         DN.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
         DN.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
         DN.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
         DN.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
         DN.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
         DN.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
         DN.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
         DN.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
         DN.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
         DN.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
         DN.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
         DN.f[d000] = &DD[d000 * numberOfLBnodes];
         DN.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
         DN.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
         DN.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
         DN.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
         DN.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
         DN.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
         DN.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
         DN.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //directions allways exchange
	  //(-1 -1 -1) (-1  0  0) ( 0 -1  0) ( 0  0 -1) (-1 -1  0) (-1  0 -1) ( 0 -1 -1) ( 1  1 -1) ( 1 -1  1) (-1  1  1) ( 1 -1  0) ( 1  0 -1) ( 0  1 -1)
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //directions exchange if solid neighbor
	  //( 1  1  1) ( 1  0  0) ( 0  1  0) ( 0  0  1) ( 1  1  0) ( 1  0  1) ( 0  1  1) (-1 -1  1) (-1  1 -1) ( 1 -1 -1) (-1  1  0) (-1  0  1) ( 0 -1  1)
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real q, tmp;
      q = q_dirE[k];   if (q>=c0o1 && q<=c1o1){ if (geom[kw  ] < GEO_FLUID){tmp = (DN.f[dM00])[kw  ]; (DN.f[dM00])[kw  ]=(D.f[dM00])[kw  ]; (D.f[dM00])[kw  ]=tmp;}}
	  q = q_dirW[k];   if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[dP00])[ke  ]; (DN.f[dP00])[ke  ]=(D.f[dP00])[ke  ]; (D.f[dP00])[ke  ]=tmp;}}
      q = q_dirN[k];   if (q>=c0o1 && q<=c1o1){ if (geom[ks  ] < GEO_FLUID){tmp = (DN.f[DIR_0M0])[ks  ]; (DN.f[DIR_0M0])[ks  ]=(D.f[DIR_0M0])[ks  ]; (D.f[DIR_0M0])[ks  ]=tmp;}}
      q = q_dirS[k];   if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_0P0])[kn  ]; (DN.f[DIR_0P0])[kn  ]=(D.f[DIR_0P0])[kn  ]; (D.f[DIR_0P0])[kn  ]=tmp;}}
      q = q_dirT[k];   if (q>=c0o1 && q<=c1o1){ if (geom[kb  ] < GEO_FLUID){tmp = (DN.f[DIR_00M])[kb  ]; (DN.f[DIR_00M])[kb  ]=(D.f[DIR_00M])[kb  ]; (D.f[DIR_00M])[kb  ]=tmp;}}
      q = q_dirB[k];   if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_00P])[kt  ]; (DN.f[DIR_00P])[kt  ]=(D.f[DIR_00P])[kt  ]; (D.f[DIR_00P])[kt  ]=tmp;}}
      q = q_dirNE[k];  if (q>=c0o1 && q<=c1o1){ if (geom[ksw ] < GEO_FLUID){tmp = (DN.f[DIR_MM0])[ksw ]; (DN.f[DIR_MM0])[ksw ]=(D.f[DIR_MM0])[ksw ]; (D.f[DIR_MM0])[ksw ]=tmp;}}
      q = q_dirSW[k];  if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_PP0])[kne ]; (DN.f[DIR_PP0])[kne ]=(D.f[DIR_PP0])[kne ]; (D.f[DIR_PP0])[kne ]=tmp;}}
      q = q_dirSE[k];  if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_MP0])[knw ]; (DN.f[DIR_MP0])[knw ]=(D.f[DIR_MP0])[knw ]; (D.f[DIR_MP0])[knw ]=tmp;}}
      q = q_dirNW[k];  if (q>=c0o1 && q<=c1o1){ if (geom[kmp0] < GEO_FLUID){tmp = (DN.f[DIR_PM0])[kse ]; (DN.f[DIR_PM0])[kse ]=(D.f[DIR_PM0])[kse ]; (D.f[DIR_PM0])[kse ]=tmp;}}
      q = q_dirTE[k];  if (q>=c0o1 && q<=c1o1){ if (geom[kbw ] < GEO_FLUID){tmp = (DN.f[DIR_M0M])[kbw ]; (DN.f[DIR_M0M])[kbw ]=(D.f[DIR_M0M])[kbw ]; (D.f[DIR_M0M])[kbw ]=tmp;}}
      q = q_dirBW[k];  if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_P0P])[kte ]; (DN.f[DIR_P0P])[kte ]=(D.f[DIR_P0P])[kte ]; (D.f[DIR_P0P])[kte ]=tmp;}}
      q = q_dirBE[k];  if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_M0P])[ktw ]; (DN.f[DIR_M0P])[ktw ]=(D.f[DIR_M0P])[ktw ]; (D.f[DIR_M0P])[ktw ]=tmp;}}
      q = q_dirTW[k];  if (q>=c0o1 && q<=c1o1){ if (geom[km0p] < GEO_FLUID){tmp = (DN.f[DIR_P0M])[kbe ]; (DN.f[DIR_P0M])[kbe ]=(D.f[DIR_P0M])[kbe ]; (D.f[DIR_P0M])[kbe ]=tmp;}}
      q = q_dirTN[k];  if (q>=c0o1 && q<=c1o1){ if (geom[kbs ] < GEO_FLUID){tmp = (DN.f[DIR_0MM])[kbs ]; (DN.f[DIR_0MM])[kbs ]=(D.f[DIR_0MM])[kbs ]; (D.f[DIR_0MM])[kbs ]=tmp;}}
      q = q_dirBS[k];  if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_0PP])[ktn ]; (DN.f[DIR_0PP])[ktn ]=(D.f[DIR_0PP])[ktn ]; (D.f[DIR_0PP])[ktn ]=tmp;}}
      q = q_dirBN[k];  if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_0MP])[kts ]; (DN.f[DIR_0MP])[kts ]=(D.f[DIR_0MP])[kts ]; (D.f[DIR_0MP])[kts ]=tmp;}}
      q = q_dirTS[k];  if (q>=c0o1 && q<=c1o1){ if (geom[k0mp] < GEO_FLUID){tmp = (DN.f[DIR_0PM])[kbn ]; (DN.f[DIR_0PM])[kbn ]=(D.f[DIR_0PM])[kbn ]; (D.f[DIR_0PM])[kbn ]=tmp;}}
      q = q_dirTNE[k]; if (q>=c0o1 && q<=c1o1){ if (geom[kbsw] < GEO_FLUID){tmp = (DN.f[DIR_MMM])[kbsw]; (DN.f[DIR_MMM])[kbsw]=(D.f[DIR_MMM])[kbsw]; (D.f[DIR_MMM])[kbsw]=tmp;}}
      q = q_dirBSW[k]; if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_PPP])[ktne]; (DN.f[DIR_PPP])[ktne]=(D.f[DIR_PPP])[ktne]; (D.f[DIR_PPP])[ktne]=tmp;}}
      q = q_dirBNE[k]; if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_MMP])[ktsw]; (DN.f[DIR_MMP])[ktsw]=(D.f[DIR_MMP])[ktsw]; (D.f[DIR_MMP])[ktsw]=tmp;}}
      q = q_dirTSW[k]; if (q>=c0o1 && q<=c1o1){ if (geom[kmmp] < GEO_FLUID){tmp = (DN.f[DIR_PPM])[kbne]; (DN.f[DIR_PPM])[kbne]=(D.f[DIR_PPM])[kbne]; (D.f[DIR_PPM])[kbne]=tmp;}}
      q = q_dirTSE[k]; if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_MPM])[kbnw]; (DN.f[DIR_MPM])[kbnw]=(D.f[DIR_MPM])[kbnw]; (D.f[DIR_MPM])[kbnw]=tmp;}}
      q = q_dirBNW[k]; if (q>=c0o1 && q<=c1o1){ if (geom[kmpm] < GEO_FLUID){tmp = (DN.f[DIR_PMP])[ktse]; (DN.f[DIR_PMP])[ktse]=(D.f[DIR_PMP])[ktse]; (D.f[DIR_PMP])[ktse]=tmp;}}
      q = q_dirBSE[k]; if (q>=c0o1 && q<=c1o1){ if (geom[kpmm] < GEO_FLUID){tmp = (DN.f[DIR_MPP])[ktnw]; (DN.f[DIR_MPP])[ktnw]=(D.f[DIR_MPP])[ktnw]; (D.f[DIR_MPP])[ktnw]=tmp;}}
      q = q_dirTNW[k]; if (q>=c0o1 && q<=c1o1){                            {tmp = (DN.f[DIR_PMM])[kbse]; (DN.f[DIR_PMM])[kbse]=(D.f[DIR_PMM])[kbse]; (D.f[DIR_PMM])[kbse]=tmp;}}
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
