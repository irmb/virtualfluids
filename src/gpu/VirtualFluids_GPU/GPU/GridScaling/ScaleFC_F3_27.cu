#include "hip/hip_runtime.h"
//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ / 
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
/* Device code */
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;

//////////////////////////////////////////////////////////////////////////
__global__ void scaleFC_comp_D3Q27F3_2018(real* DC,
													 real* DF,
													 real* G6,
													 unsigned int* neighborCX,
													 unsigned int* neighborCY,
													 unsigned int* neighborCZ,
													 unsigned int* neighborFX,
													 unsigned int* neighborFY,
													 unsigned int* neighborFZ,
													 unsigned long long numberOfLBnodesCoarse, 
													 unsigned long long numberOfLBnodesFine, 
													 bool isEvenTimestep,
													 unsigned int* posC, 
													 unsigned int* posFSWB, 
													 unsigned int kFC, 
													 real omCoarse, 
													 real omFine, 
													 real nu, 
													 unsigned int nxC, 
													 unsigned int nyC, 
													 unsigned int nxF, 
													 unsigned int nyF,
													 ICellNeighFC offFC)
{
   real 
	   *fP00source, *fM00source, *f0P0source, *f0M0source, *f00Psource, *f00Msource, *fPP0source, *fMM0source, *fPM0source,
	   *fMP0source, *fP0Psource, *fM0Msource, *fP0Msource, *fM0Psource, *f0PPsource, *f0MMsource, *f0PMsource, *f0MPsource,
	   *f000source, *fMMMsource, *fMMPsource, *fMPPsource, *fMPMsource, *fPPMsource, *fPPPsource, *fPMPsource, *fPMMsource;


   fP00source = &DF[DIR_P00 * numberOfLBnodesFine];
   fM00source = &DF[DIR_M00 * numberOfLBnodesFine];
   f0P0source = &DF[DIR_0P0 * numberOfLBnodesFine];
   f0M0source = &DF[DIR_0M0 * numberOfLBnodesFine];
   f00Psource = &DF[DIR_00P * numberOfLBnodesFine];
   f00Msource = &DF[DIR_00M * numberOfLBnodesFine];
   fPP0source = &DF[DIR_PP0 * numberOfLBnodesFine];
   fMM0source = &DF[DIR_MM0 * numberOfLBnodesFine];
   fPM0source = &DF[DIR_PM0 * numberOfLBnodesFine];
   fMP0source = &DF[DIR_MP0 * numberOfLBnodesFine];
   fP0Psource = &DF[DIR_P0P * numberOfLBnodesFine];
   fM0Msource = &DF[DIR_M0M * numberOfLBnodesFine];
   fP0Msource = &DF[DIR_P0M * numberOfLBnodesFine];
   fM0Psource = &DF[DIR_M0P * numberOfLBnodesFine];
   f0PPsource = &DF[DIR_0PP * numberOfLBnodesFine];
   f0MMsource = &DF[DIR_0MM * numberOfLBnodesFine];
   f0PMsource = &DF[DIR_0PM * numberOfLBnodesFine];
   f0MPsource = &DF[DIR_0MP * numberOfLBnodesFine];
   f000source = &DF[DIR_000 * numberOfLBnodesFine];
   fMMMsource = &DF[DIR_MMM * numberOfLBnodesFine];
   fMMPsource = &DF[DIR_MMP * numberOfLBnodesFine];
   fMPPsource = &DF[DIR_MPP * numberOfLBnodesFine];
   fMPMsource = &DF[DIR_MPM * numberOfLBnodesFine];
   fPPMsource = &DF[DIR_PPM * numberOfLBnodesFine];
   fPPPsource = &DF[DIR_PPP * numberOfLBnodesFine];
   fPMPsource = &DF[DIR_PMP * numberOfLBnodesFine];
   fPMMsource = &DF[DIR_PMM * numberOfLBnodesFine];

   real
	   *fP00dest, *fM00dest, *f0P0dest, *f0M0dest, *f00Pdest, *f00Mdest, *fPP0dest, *fMM0dest, *fPM0dest,
	   *fMP0dest, *fP0Pdest, *fM0Mdest, *fP0Mdest, *fM0Pdest, *f0PPdest, *f0MMdest, *f0PMdest, *f0MPdest,
	   *f000dest, *fMMMdest, *fMMPdest, *fMPPdest, *fMPMdest, *fPPMdest, *fPPPdest, *fPMPdest, *fPMMdest;

   if (isEvenTimestep==true)
   {
	   fP00dest = &DC[DIR_P00 * numberOfLBnodesCoarse];
	   fM00dest = &DC[DIR_M00 * numberOfLBnodesCoarse];
	   f0P0dest = &DC[DIR_0P0 * numberOfLBnodesCoarse];
	   f0M0dest = &DC[DIR_0M0 * numberOfLBnodesCoarse];
	   f00Pdest = &DC[DIR_00P * numberOfLBnodesCoarse];
	   f00Mdest = &DC[DIR_00M * numberOfLBnodesCoarse];
	   fPP0dest = &DC[DIR_PP0 * numberOfLBnodesCoarse];
	   fMM0dest = &DC[DIR_MM0 * numberOfLBnodesCoarse];
	   fPM0dest = &DC[DIR_PM0 * numberOfLBnodesCoarse];
	   fMP0dest = &DC[DIR_MP0 * numberOfLBnodesCoarse];
	   fP0Pdest = &DC[DIR_P0P * numberOfLBnodesCoarse];
	   fM0Mdest = &DC[DIR_M0M * numberOfLBnodesCoarse];
	   fP0Mdest = &DC[DIR_P0M * numberOfLBnodesCoarse];
	   fM0Pdest = &DC[DIR_M0P * numberOfLBnodesCoarse];
	   f0PPdest = &DC[DIR_0PP * numberOfLBnodesCoarse];
	   f0MMdest = &DC[DIR_0MM * numberOfLBnodesCoarse];
	   f0PMdest = &DC[DIR_0PM * numberOfLBnodesCoarse];
	   f0MPdest = &DC[DIR_0MP * numberOfLBnodesCoarse];
	   f000dest = &DC[DIR_000 * numberOfLBnodesCoarse];
	   fMMMdest = &DC[DIR_MMM * numberOfLBnodesCoarse];
	   fMMPdest = &DC[DIR_MMP * numberOfLBnodesCoarse];
	   fMPPdest = &DC[DIR_MPP * numberOfLBnodesCoarse];
	   fMPMdest = &DC[DIR_MPM * numberOfLBnodesCoarse];
	   fPPMdest = &DC[DIR_PPM * numberOfLBnodesCoarse];
	   fPPPdest = &DC[DIR_PPP * numberOfLBnodesCoarse];
	   fPMPdest = &DC[DIR_PMP * numberOfLBnodesCoarse];
	   fPMMdest = &DC[DIR_PMM * numberOfLBnodesCoarse];
   } 
   else
   {
	   fP00dest = &DC[DIR_M00 * numberOfLBnodesCoarse];
	   fM00dest = &DC[DIR_P00 * numberOfLBnodesCoarse];
	   f0P0dest = &DC[DIR_0M0 * numberOfLBnodesCoarse];
	   f0M0dest = &DC[DIR_0P0 * numberOfLBnodesCoarse];
	   f00Pdest = &DC[DIR_00M * numberOfLBnodesCoarse];
	   f00Mdest = &DC[DIR_00P * numberOfLBnodesCoarse];
	   fPP0dest = &DC[DIR_MM0 * numberOfLBnodesCoarse];
	   fMM0dest = &DC[DIR_PP0 * numberOfLBnodesCoarse];
	   fPM0dest = &DC[DIR_MP0 * numberOfLBnodesCoarse];
	   fMP0dest = &DC[DIR_PM0 * numberOfLBnodesCoarse];
	   fP0Pdest = &DC[DIR_M0M * numberOfLBnodesCoarse];
	   fM0Mdest = &DC[DIR_P0P * numberOfLBnodesCoarse];
	   fP0Mdest = &DC[DIR_M0P * numberOfLBnodesCoarse];
	   fM0Pdest = &DC[DIR_P0M * numberOfLBnodesCoarse];
	   f0PPdest = &DC[DIR_0MM * numberOfLBnodesCoarse];
	   f0MMdest = &DC[DIR_0PP * numberOfLBnodesCoarse];
	   f0PMdest = &DC[DIR_0MP * numberOfLBnodesCoarse];
	   f0MPdest = &DC[DIR_0PM * numberOfLBnodesCoarse];
	   f000dest = &DC[DIR_000 * numberOfLBnodesCoarse];
	   fMMMdest = &DC[DIR_PPP * numberOfLBnodesCoarse];
	   fMMPdest = &DC[DIR_PPM * numberOfLBnodesCoarse];
	   fMPPdest = &DC[DIR_PMM * numberOfLBnodesCoarse];
	   fMPMdest = &DC[DIR_PMP * numberOfLBnodesCoarse];
	   fPPMdest = &DC[DIR_MMP * numberOfLBnodesCoarse];
	   fPPPdest = &DC[DIR_MMM * numberOfLBnodesCoarse];
	   fPMPdest = &DC[DIR_MPM * numberOfLBnodesCoarse];
	   fPMMdest = &DC[DIR_MPP * numberOfLBnodesCoarse];
   }

   Distributions6 G;
   if (isEvenTimestep == true)
   {
	   G.g[DIR_P00] = &G6[DIR_P00 * numberOfLBnodesCoarse];
	   G.g[DIR_M00] = &G6[DIR_M00 * numberOfLBnodesCoarse];
	   G.g[DIR_0P0] = &G6[DIR_0P0 * numberOfLBnodesCoarse];
	   G.g[DIR_0M0] = &G6[DIR_0M0 * numberOfLBnodesCoarse];
	   G.g[DIR_00P] = &G6[DIR_00P * numberOfLBnodesCoarse];
	   G.g[DIR_00M] = &G6[DIR_00M * numberOfLBnodesCoarse];
   }
   else
   {
	   G.g[DIR_M00] = &G6[DIR_P00 * numberOfLBnodesCoarse];
	   G.g[DIR_P00] = &G6[DIR_M00 * numberOfLBnodesCoarse];
	   G.g[DIR_0M0] = &G6[DIR_0P0 * numberOfLBnodesCoarse];
	   G.g[DIR_0P0] = &G6[DIR_0M0 * numberOfLBnodesCoarse];
	   G.g[DIR_00M] = &G6[DIR_00P * numberOfLBnodesCoarse];
	   G.g[DIR_00P] = &G6[DIR_00M * numberOfLBnodesCoarse];
   }

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  ix = threadIdx.x;  // Globaler x-Index 
   const unsigned  iy = blockIdx.x;   // Globaler y-Index 
   const unsigned  iz = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*iz + iy) + ix;
   //////////////////////////////////////////////////////////////////////////

   ////////////////////////////////////////////////////////////////////////////////
   real eps_new = c2o1;
   real omegaS = omFine;//-omFine;
   real o  = omCoarse;//-omCoarse;
   //real op = one;
   //real cu_sq;

   real xoff,    yoff,    zoff;
   real xoff_sq, yoff_sq, zoff_sq;

   // real drho;
   real        vvx, vvy, vvz, vx2, vy2, vz2;
   real        press;//,drho,vx1,vx2,vx3;
   real        /*pressMMM,*/drhoMMM,vx1MMM,vx2MMM,vx3MMM;
   real        /*pressMMP,*/drhoMMP,vx1MMP,vx2MMP,vx3MMP;
   real        /*pressMPP,*/drhoMPP,vx1MPP,vx2MPP,vx3MPP;
   real        /*pressMPM,*/drhoMPM,vx1MPM,vx2MPM,vx3MPM;
   real        /*pressPPM,*/drhoPPM,vx1PPM,vx2PPM,vx3PPM;
   real        /*pressPPP,*/drhoPPP,vx1PPP,vx2PPP,vx3PPP;
   real        /*pressPMP,*/drhoPMP,vx1PMP,vx2PMP,vx3PMP;
   real        /*pressPMM,*/drhoPMM,vx1PMM,vx2PMM,vx3PMM;
   real        fP00,fM00,f0P0,f0M0,f00P,f00M,fPP0,fMM0,fPM0,fMP0,fP0P,fM0M,fP0M,fM0P,f0PP,f0MM,f0PM,f0MP,f000,fPPP, fMMP, fPMP, fMPP, fPPM, fMMM, fPMM, fMPM;
   real        kxyFromfcNEQMMM, kyzFromfcNEQMMM, kxzFromfcNEQMMM, kxxMyyFromfcNEQMMM, kxxMzzFromfcNEQMMM, kyyMzzFromfcNEQMMM;
   real        kxyFromfcNEQMMP, kyzFromfcNEQMMP, kxzFromfcNEQMMP, kxxMyyFromfcNEQMMP, kxxMzzFromfcNEQMMP, kyyMzzFromfcNEQMMP;
   real        kxyFromfcNEQMPP, kyzFromfcNEQMPP, kxzFromfcNEQMPP, kxxMyyFromfcNEQMPP, kxxMzzFromfcNEQMPP, kyyMzzFromfcNEQMPP;
   real        kxyFromfcNEQMPM, kyzFromfcNEQMPM, kxzFromfcNEQMPM, kxxMyyFromfcNEQMPM, kxxMzzFromfcNEQMPM, kyyMzzFromfcNEQMPM;
   real        kxyFromfcNEQPPM, kyzFromfcNEQPPM, kxzFromfcNEQPPM, kxxMyyFromfcNEQPPM, kxxMzzFromfcNEQPPM, kyyMzzFromfcNEQPPM;
   real        kxyFromfcNEQPPP, kyzFromfcNEQPPP, kxzFromfcNEQPPP, kxxMyyFromfcNEQPPP, kxxMzzFromfcNEQPPP, kyyMzzFromfcNEQPPP;
   real        kxyFromfcNEQPMP, kyzFromfcNEQPMP, kxzFromfcNEQPMP, kxxMyyFromfcNEQPMP, kxxMzzFromfcNEQPMP, kyyMzzFromfcNEQPMP;
   real        kxyFromfcNEQPMM, kyzFromfcNEQPMM, kxzFromfcNEQPMM, kxxMyyFromfcNEQPMM, kxxMzzFromfcNEQPMM, kyyMzzFromfcNEQPMM;
   real        a0, ax, ay, az, axx, ayy, azz, axy, axz, ayz, b0, bx, by, bz, bxx, byy, bzz, bxy, bxz, byz, c0, cx, cy, cz, cxx, cyy, czz, cxy, cxz, cyz/*, axyz, bxyz, cxyz*/;
   real        d0, dx, dy, dz, dxy, dxz, dyz/*, dxyz*/;

   if(k<kFC)
   {
      //////////////////////////////////////////////////////////////////////////
      xoff = offFC.x[k];
      yoff = offFC.y[k];
      zoff = offFC.z[k];      
      xoff_sq = xoff * xoff;
      yoff_sq = yoff * yoff;
      zoff_sq = zoff * zoff;
      //////////////////////////////////////////////////////////////////////////
      //SWB//
      //////////////////////////////////////////////////////////////////////////
      //index 0
      unsigned int k000base = posFSWB[k];
      unsigned int kM00base = neighborFX[k000base];
      unsigned int k0M0base = neighborFY[k000base];
      unsigned int k00Mbase = neighborFZ[k000base];
      unsigned int kMM0base = neighborFY[kM00base];
      unsigned int kM0Mbase = neighborFZ[kM00base];
      unsigned int k0MMbase = neighborFZ[k0M0base];
      unsigned int kMMMbase = neighborFZ[kMM0base];
      //////////////////////////////////////////////////////////////////////////
      //index 
      unsigned int k000 = k000base;
      unsigned int kM00 = kM00base;   
      unsigned int k0M0 = k0M0base;   
      unsigned int k00M = k00Mbase;   
      unsigned int kMM0 = kMM0base;  
      unsigned int kM0M = kM0Mbase;  
      unsigned int k0MM = k0MMbase;  
      unsigned int kMMM = kMMMbase; 
      ////////////////////////////////////////////////////////////////////////////////
      fP00 = fP00source[k000];
      fM00 = fM00source[kM00];
      f0P0 = f0P0source[k000];
      f0M0 = f0M0source[k0M0];
      f00P = f00Psource[k000];
      f00M = f00Msource[k00M];
      fPP0 = fPP0source[k000];
      fMM0 = fMM0source[kMM0];
      fPM0 = fPM0source[k0M0];
      fMP0 = fMP0source[kM00];
      fP0P = fP0Psource[k000];
      fM0M = fM0Msource[kM0M];
      fP0M = fP0Msource[k00M];
      fM0P = fM0Psource[kM00];
      f0PP = f0PPsource[k000];
      f0MM = f0MMsource[k0MM];
      f0PM = f0PMsource[k00M];
      f0MP = f0MPsource[k0M0];
      f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
      fPMP = fPMPsource[k0M0];
      fPMM = fPMMsource[k0MM];

      drhoMMM = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1MMM  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoMMM);
	  vx2MMM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoMMM);
	  vx3MMM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoMMM);

	  kxyFromfcNEQMMM    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoMMM) - ((vx1MMM*vx2MMM)));
	  kyzFromfcNEQMMM    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoMMM) - ((vx2MMM*vx3MMM)));
	  kxzFromfcNEQMMM    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoMMM) - ((vx1MMM*vx3MMM)));
	  kxxMyyFromfcNEQMMM = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoMMM) - ((vx1MMM*vx1MMM - vx2MMM*vx2MMM)));
	  kxxMzzFromfcNEQMMM = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoMMM) - ((vx1MMM*vx1MMM - vx3MMM*vx3MMM)));
	  kyyMzzFromfcNEQMMM = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoMMM) - ((vx2MMM*vx2MMM - vx3MMM*vx3MMM)));

      //////////////////////////////////////////////////////////////////////////
      //SWT//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = k00M;
      kM00 = kM0M;   
      k0M0 = k0MM;   
      k00M = neighborFZ[k00M];   
      kMM0 = kMMM;  
      kM0M = neighborFZ[kM0M];  
      k0MM = neighborFZ[k0MM];  
      kMMM = neighborFZ[kMMM]; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoMMP = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1MMP  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoMMP);
	  vx2MMP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoMMP);
	  vx3MMP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoMMP);

	  kxyFromfcNEQMMP    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoMMP) - ((vx1MMP*vx2MMP)));
	  kyzFromfcNEQMMP    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoMMP) - ((vx2MMP*vx3MMP)));
	  kxzFromfcNEQMMP    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoMMP) - ((vx1MMP*vx3MMP)));
	  kxxMyyFromfcNEQMMP = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoMMP) - ((vx1MMP*vx1MMP - vx2MMP*vx2MMP)));
	  kxxMzzFromfcNEQMMP = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoMMP) - ((vx1MMP*vx1MMP - vx3MMP*vx3MMP)));
	  kyyMzzFromfcNEQMMP = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoMMP) - ((vx2MMP*vx2MMP - vx3MMP*vx3MMP)));

      //////////////////////////////////////////////////////////////////////////
      //SET//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = kM00;
      kM00 = neighborFX[kM00];   
      k0M0 = kMM0;   
      k00M = kM0M;   
      kMM0 = neighborFX[kMM0];  
      kM0M = neighborFX[kM0M];  
      k0MM = kMMM;  
      kMMM = neighborFX[kMMM]; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoPMP = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1PMP  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoPMP);
	  vx2PMP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoPMP);
	  vx3PMP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoPMP);

	  kxyFromfcNEQPMP    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoPMP) - ((vx1PMP*vx2PMP)));
	  kyzFromfcNEQPMP    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoPMP) - ((vx2PMP*vx3PMP)));
	  kxzFromfcNEQPMP    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoPMP) - ((vx1PMP*vx3PMP)));
	  kxxMyyFromfcNEQPMP = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoPMP) - ((vx1PMP*vx1PMP - vx2PMP*vx2PMP)));
	  kxxMzzFromfcNEQPMP = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoPMP) - ((vx1PMP*vx1PMP - vx3PMP*vx3PMP)));
	  kyyMzzFromfcNEQPMP = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoPMP) - ((vx2PMP*vx2PMP - vx3PMP*vx3PMP)));

      //////////////////////////////////////////////////////////////////////////
      //SEB//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k00M = k000;   
      kM0M = kM00;  
      k0MM = k0M0;  
      kMMM = kMM0; 
      k000 = kM00base;
      kM00 = neighborFX[kM00base];   
      k0M0 = kMM0base;   
      kMM0 = neighborFX[kMM0base];  
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoPMM = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1PMM  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoPMM);
	  vx2PMM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoPMM);
	  vx3PMM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoPMM);

	  kxyFromfcNEQPMM    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoPMM) - ((vx1PMM*vx2PMM)));
	  kyzFromfcNEQPMM    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoPMM) - ((vx2PMM*vx3PMM)));
	  kxzFromfcNEQPMM    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoPMM) - ((vx1PMM*vx3PMM)));
	  kxxMyyFromfcNEQPMM = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoPMM) - ((vx1PMM*vx1PMM - vx2PMM*vx2PMM)));
	  kxxMzzFromfcNEQPMM = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoPMM) - ((vx1PMM*vx1PMM - vx3PMM*vx3PMM)));
	  kyyMzzFromfcNEQPMM = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoPMM) - ((vx2PMM*vx2PMM - vx3PMM*vx3PMM)));

      //////////////////////////////////////////////////////////////////////////
      //NWB//
      //////////////////////////////////////////////////////////////////////////
      //index 0
      k000base = k0M0base;
      kM00base = kMM0base;
      k0M0base = neighborFY[k0M0base];
      k00Mbase = k0MMbase;
      kMM0base = neighborFY[kMM0base];
      kM0Mbase = kMMMbase;
      k0MMbase = neighborFY[k0MMbase];
      kMMMbase = neighborFY[kMMMbase];
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = k000base;
      kM00 = kM00base;   
      k0M0 = k0M0base;   
      k00M = k00Mbase;   
      kMM0 = kMM0base;  
      kM0M = kM0Mbase;  
      k0MM = k0MMbase;  
      kMMM = kMMMbase; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoMPM = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1MPM  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoMPM);
	  vx2MPM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoMPM);
	  vx3MPM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoMPM);

	  kxyFromfcNEQMPM    = -c3o1*omegaS*   ((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoMPM) - ((vx1MPM*vx2MPM)));
	  kyzFromfcNEQMPM    = -c3o1*omegaS*   ((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoMPM) - ((vx2MPM*vx3MPM)));
	  kxzFromfcNEQMPM    = -c3o1*omegaS*   ((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoMPM) - ((vx1MPM*vx3MPM)));
	  kxxMyyFromfcNEQMPM = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoMPM) - ((vx1MPM*vx1MPM - vx2MPM*vx2MPM)));
	  kxxMzzFromfcNEQMPM = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoMPM) - ((vx1MPM*vx1MPM - vx3MPM*vx3MPM)));
	  kyyMzzFromfcNEQMPM = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoMPM) - ((vx2MPM*vx2MPM - vx3MPM*vx3MPM)));

	  //////////////////////////////////////////////////////////////////////////
      //NWT//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = k00M;
      kM00 = kM0M;   
      k0M0 = k0MM;   
      k00M = neighborFZ[k00M];   
      kMM0 = kMMM;  
      kM0M = neighborFZ[kM0M];  
      k0MM = neighborFZ[k0MM];  
      kMMM = neighborFZ[kMMM]; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoMPP = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1MPP  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoMPP);
	  vx2MPP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoMPP);
	  vx3MPP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoMPP);

	  kxyFromfcNEQMPP    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoMPP) - ((vx1MPP*vx2MPP)));
	  kyzFromfcNEQMPP    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoMPP) - ((vx2MPP*vx3MPP)));
	  kxzFromfcNEQMPP    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoMPP) - ((vx1MPP*vx3MPP)));
	  kxxMyyFromfcNEQMPP = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoMPP) - ((vx1MPP*vx1MPP - vx2MPP*vx2MPP)));
	  kxxMzzFromfcNEQMPP = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoMPP) - ((vx1MPP*vx1MPP - vx3MPP*vx3MPP)));
	  kyyMzzFromfcNEQMPP = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoMPP) - ((vx2MPP*vx2MPP - vx3MPP*vx3MPP)));

      //////////////////////////////////////////////////////////////////////////
      //NET//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = kM00;
      kM00 = neighborFX[kM00];   
      k0M0 = kMM0;   
      k00M = kM0M;   
      kMM0 = neighborFX[kMM0];  
      kM0M = neighborFX[kM0M];  
      k0MM = kMMM;  
      kMMM = neighborFX[kMMM]; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoPPP = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1PPP  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoPPP);
	  vx2PPP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoPPP);
	  vx3PPP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoPPP);

	  kxyFromfcNEQPPP    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoPPP) - ((vx1PPP*vx2PPP)));
	  kyzFromfcNEQPPP    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoPPP) - ((vx2PPP*vx3PPP)));
	  kxzFromfcNEQPPP    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoPPP) - ((vx1PPP*vx3PPP)));
	  kxxMyyFromfcNEQPPP = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoPPP) - ((vx1PPP*vx1PPP - vx2PPP*vx2PPP)));
	  kxxMzzFromfcNEQPPP = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoPPP) - ((vx1PPP*vx1PPP - vx3PPP*vx3PPP)));
	  kyyMzzFromfcNEQPPP = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoPPP) - ((vx2PPP*vx2PPP - vx3PPP*vx3PPP)));

      //////////////////////////////////////////////////////////////////////////
      //NEB//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k00M = k000;   
      kM0M = kM00;  
      k0MM = k0M0;  
      kMMM = kMM0; 
      k000 = kM00base;
      kM00 = neighborFX[kM00base];   
      k0M0 = kMM0base;   
      kMM0 = neighborFX[kMM0base];  
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoPPM = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1PPM  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoPPM);
	  vx2PPM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoPPM);
	  vx3PPM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoPPM);

	  kxyFromfcNEQPPM    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoPPM) - ((vx1PPM*vx2PPM)));
	  kyzFromfcNEQPPM    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoPPM) - ((vx2PPM*vx3PPM)));
	  kxzFromfcNEQPPM    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoPPM) - ((vx1PPM*vx3PPM)));
	  kxxMyyFromfcNEQPPM = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoPPM) - ((vx1PPM*vx1PPM - vx2PPM*vx2PPM)));
	  kxxMzzFromfcNEQPPM = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoPPM) - ((vx1PPM*vx1PPM - vx3PPM*vx3PPM)));
	  kyyMzzFromfcNEQPPM = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoPPM) - ((vx2PPM*vx2PPM - vx3PPM*vx3PPM)));

      //////////////////////////////////////////////////////////////////////////
      //3
      //////////////////////////////////////////////////////////////////////////
	  a0  = c1o8*(((vx1PPM + vx1MMP) + (vx1MPM + vx1PMP)) + ((vx1PMM + vx1MPP) + (vx1MMM + vx1PPP)));
	  ax  = c1o4*(((vx1PPM - vx1MMP) + (vx1PPP - vx1MMM)) + ((vx1PMP - vx1MPM) + (vx1PMM - vx1MPP)));
	  ay  = c1o4*(((vx1PPM - vx1MMP) + (vx1PPP - vx1MMM)) + ((vx1MPM - vx1PMP) + (vx1MPP - vx1PMM)));
	  az  = c1o4*(((vx1MMP - vx1PPM) + (vx1PPP - vx1MMM)) + ((vx1PMP - vx1MPM) + (vx1MPP - vx1PMM)));
	  axy = c1o2*(((vx1PPM - vx1PMP) + (vx1MMM - vx1MPP)) + ((vx1MMP - vx1MPM) + (vx1PPP - vx1PMM)));
	  axz = c1o2*(((vx1PMP - vx1PPM) + (vx1MMM - vx1MPP)) + ((vx1MPM - vx1MMP) + (vx1PPP - vx1PMM)));
	  ayz = c1o2*(((vx1PPP - vx1MPM) + (vx1PMM - vx1MMP)) + ((vx1MPP - vx1PPM) + (vx1MMM - vx1PMP)));
	  //axyz=		  ((vx1MMP - vx1PPM) + (vx1PPP - vx1MMM)) + ((vx1MPM - vx1PMP) + (vx1PMM - vx1MPP)) ;
	  b0  = c1o8*(((vx2PPM + vx2MMP) + (vx2MPM + vx2PMP)) + ((vx2PMM + vx2MPP) + (vx2MMM + vx2PPP)));
	  bx  = c1o4*(((vx2PPM - vx2MMP) + (vx2PPP - vx2MMM)) + ((vx2PMP - vx2MPM) + (vx2PMM - vx2MPP)));
	  by  = c1o4*(((vx2PPM - vx2MMP) + (vx2PPP - vx2MMM)) + ((vx2MPM - vx2PMP) + (vx2MPP - vx2PMM)));
	  bz  = c1o4*(((vx2MMP - vx2PPM) + (vx2PPP - vx2MMM)) + ((vx2PMP - vx2MPM) + (vx2MPP - vx2PMM)));
	  bxy = c1o2*(((vx2PPM - vx2MPP) + (vx2MMM - vx2PMP)) + ((vx2MMP - vx2PMM) + (vx2PPP - vx2MPM)));
	  bxz = c1o2*(((vx2MMM - vx2PPM) + (vx2PMP - vx2MPP)) + ((vx2MPM - vx2PMM) + (vx2PPP - vx2MMP)));
	  byz = c1o2*(((vx2MPP - vx2PPM) + (vx2MMM - vx2PMP)) + ((vx2PMM - vx2MMP) + (vx2PPP - vx2MPM)));
	  //bxyz=		  ((vx2MMP - vx2PPM) + (vx2PPP - vx2MMM)) + ((vx2MPM - vx2PMP) + (vx2PMM - vx2MPP)) ;
	  c0  = c1o8*(((vx3PPM + vx3MMP) + (vx3MPM + vx3PMP)) + ((vx3PMM + vx3MPP) + (vx3MMM + vx3PPP)));
	  cx  = c1o4*(((vx3PPM - vx3MMP) + (vx3PPP - vx3MMM)) + ((vx3PMP - vx3MPM) + (vx3PMM - vx3MPP)));
	  cy  = c1o4*(((vx3PPM - vx3MMP) + (vx3PPP - vx3MMM)) + ((vx3MPM - vx3PMP) + (vx3MPP - vx3PMM)));
	  cz  = c1o4*(((vx3MMP - vx3PPM) + (vx3PPP - vx3MMM)) + ((vx3PMP - vx3MPM) + (vx3MPP - vx3PMM)));
	  cxy = c1o2*(((vx3PPM - vx3PMP) + (vx3MMM - vx3MPP)) + ((vx3MMP - vx3MPM) + (vx3PPP - vx3PMM)));
	  cxz = c1o2*(((vx3MMM - vx3PPM) + (vx3PMP - vx3MPP)) + ((vx3MPM - vx3PMM) + (vx3PPP - vx3MMP)));
	  cyz = c1o2*(((vx3MMM - vx3PPM) + (vx3MPP - vx3PMP)) + ((vx3PMM - vx3MPM) + (vx3PPP - vx3MMP)));
	  //cxyz=		  ((vx3MMP - vx3PPM) + (vx3PPP - vx3MMM)) + ((vx3MPM - vx3PMP) + (vx3PMM - vx3MPP)) ;

	  //bxx = zero;
	  //cxx = zero;
	  //ayy = zero;
	  //cyy = zero;
	  //azz = zero;
	  //bzz = zero;
	  //axx = zero;
	  //byy = zero;
	  //czz = zero;

	  bxx = c1o8*(((kxyFromfcNEQPPP - kxyFromfcNEQMMM) + (kxyFromfcNEQPMM - kxyFromfcNEQMPP)) + ((kxyFromfcNEQPMP - kxyFromfcNEQMPM) + (kxyFromfcNEQPPM - kxyFromfcNEQMMP))) - c1o2*axy;
	  cxx = c1o8*(((kxzFromfcNEQPPP - kxzFromfcNEQMMM) + (kxzFromfcNEQPMM - kxzFromfcNEQMPP)) + ((kxzFromfcNEQPMP - kxzFromfcNEQMPM) + (kxzFromfcNEQPPM - kxzFromfcNEQMMP))) - c1o2*axz;

	  ayy = c1o8*(((kxyFromfcNEQPPP - kxyFromfcNEQMMM) + (kxyFromfcNEQPPM - kxyFromfcNEQMMP)) - ((kxyFromfcNEQPMM - kxyFromfcNEQMPP) + (kxyFromfcNEQPMP - kxyFromfcNEQMPM))) - c1o2*bxy;
	  cyy = c1o8*(((kyzFromfcNEQPPP - kyzFromfcNEQMMM) + (kyzFromfcNEQPPM - kyzFromfcNEQMMP)) - ((kyzFromfcNEQPMM - kyzFromfcNEQMPP) + (kyzFromfcNEQPMP - kyzFromfcNEQMPM))) - c1o2*byz;

	  azz = c1o8*(((kxzFromfcNEQPPP - kxzFromfcNEQMMM) - (kxzFromfcNEQPMM - kxzFromfcNEQMPP)) + ((kxzFromfcNEQPMP - kxzFromfcNEQMPM) - (kxzFromfcNEQPPM - kxzFromfcNEQMMP))) - c1o2*cxz;
	  bzz = c1o8*(((kyzFromfcNEQPPP - kyzFromfcNEQMMM) - (kyzFromfcNEQPMM - kyzFromfcNEQMPP)) + ((kyzFromfcNEQPMP - kyzFromfcNEQMPM) - (kyzFromfcNEQPPM - kyzFromfcNEQMMP))) - c1o2*cyz;

	  axx = ( c1o16*(((kxxMyyFromfcNEQPPP - kxxMyyFromfcNEQMMM) + (kxxMyyFromfcNEQPMM - kxxMyyFromfcNEQMPP)) + ((kxxMyyFromfcNEQPMP - kxxMyyFromfcNEQMPM) + (kxxMyyFromfcNEQPPM - kxxMyyFromfcNEQMMP))) - c1o4*bxy)
		  + ( c1o16*(((kxxMzzFromfcNEQPPP - kxxMzzFromfcNEQMMM) + (kxxMzzFromfcNEQPMM - kxxMzzFromfcNEQMPP)) + ((kxxMzzFromfcNEQPMP - kxxMzzFromfcNEQMPM) + (kxxMzzFromfcNEQPPM - kxxMzzFromfcNEQMMP))) - c1o4*cxz);

	  byy = (-c1o16*(((kxxMyyFromfcNEQPPP - kxxMyyFromfcNEQMMM) - (kxxMyyFromfcNEQPMM - kxxMyyFromfcNEQMPP)) + ((kxxMyyFromfcNEQPPM - kxxMyyFromfcNEQMMP) - (kxxMyyFromfcNEQPMP - kxxMyyFromfcNEQMPM))) - c1o4*axy)
		  + ( c1o16*(((kyyMzzFromfcNEQPPP - kyyMzzFromfcNEQMMM) - (kyyMzzFromfcNEQPMM - kyyMzzFromfcNEQMPP)) + ((kyyMzzFromfcNEQPPM - kyyMzzFromfcNEQMMP) - (kyyMzzFromfcNEQPMP - kyyMzzFromfcNEQMPM))) - c1o4*cyz);

	  czz = (-c1o16*(((kxxMzzFromfcNEQPPP - kxxMzzFromfcNEQMMM) - (kxxMzzFromfcNEQPMM - kxxMzzFromfcNEQMPP)) - ((kxxMzzFromfcNEQPPM - kxxMzzFromfcNEQMMP) - (kxxMzzFromfcNEQPMP - kxxMzzFromfcNEQMPM))) - c1o4*axz)
		  + ( c1o16*(((kyyMzzFromfcNEQPPP - kyyMzzFromfcNEQMMM) - (kyyMzzFromfcNEQPMM - kyyMzzFromfcNEQMPP)) - ((kyyMzzFromfcNEQPPM - kyyMzzFromfcNEQMMP) - (kyyMzzFromfcNEQPMP - kyyMzzFromfcNEQMPM))) - c1o4*byz);

	  a0 -= c1o4*(axx + ayy + azz);
	  b0 -= c1o4*(bxx + byy + bzz);
	  c0 -= c1o4*(cxx + cyy + czz);

	  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real kxyAverage = c0;
	  real kyzAverage = c0;
	  real kxzAverage = c0;
	  real kxxMyyAverage = c0;
	  real kxxMzzAverage = c0;
	  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////Press
	  //d0   = ( pressPPM + pressPPP + pressMPM + pressMPP + pressPMM + pressPMP + pressMMM + pressMMP) * c1o8;
	  //dx   = ( pressPPM + pressPPP - pressMPM - pressMPP + pressPMM + pressPMP - pressMMM - pressMMP) * c1o4;
	  //dy   = ( pressPPM + pressPPP + pressMPM + pressMPP - pressPMM - pressPMP - pressMMM - pressMMP) * c1o4;
	  //dz   = (-pressPPM + pressPPP - pressMPM + pressMPP - pressPMM + pressPMP - pressMMM + pressMMP) * c1o4;
	  //dxy  = ( pressPPM + pressPPP - pressMPM - pressMPP - pressPMM - pressPMP + pressMMM + pressMMP) * c1o2;
	  //dxz  = (-pressPPM + pressPPP + pressMPM - pressMPP - pressPMM + pressPMP + pressMMM - pressMMP) * c1o2;
	  //dyz  = (-pressPPM + pressPPP - pressMPM + pressMPP + pressPMM - pressPMP + pressMMM - pressMMP) * c1o2;
	  //dxyz =  -pressPPM + pressPPP + pressMPM - pressMPP + pressPMM - pressPMP - pressMMM + pressMMP;
	  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //drho
	  d0   = ( ((drhoPPM + drhoMMP) + (drhoMPM + drhoPMP)) + ((drhoPMM + drhoMPP) + (drhoMMM + drhoPPP))) * c1o8;
	  dx   = ( ((drhoPPM - drhoMMP) + (drhoPMM - drhoMPP)) + ((drhoPMP - drhoMPM) + (drhoPPP - drhoMMM))) * c1o4;
	  dy   = ( ((drhoPPM - drhoMMP) + (drhoMPP - drhoPMM)) + ((drhoMPM - drhoPMP) + (drhoPPP - drhoMMM))) * c1o4;
	  dz   = ( ((drhoMMP - drhoPPM) + (drhoPPP - drhoMMM)) + ((drhoPMP - drhoMPM) + (drhoMPP - drhoPMM))) * c1o4;
	  dxy  = ( ((drhoPPM - drhoPMP) + (drhoPPP - drhoPMM)) + ((drhoMMP - drhoMPM) + (drhoMMM - drhoMPP))) * c1o2;
	  dxz  = ( ((drhoMMM - drhoPPM) + (drhoPPP - drhoMMP)) + ((drhoMPM - drhoPMM) + (drhoPMP - drhoMPP))) * c1o2;
	  dyz  = ( ((drhoMPP - drhoPPM) + (drhoPPP - drhoMPM)) + ((drhoPMM - drhoMMP) + (drhoMMM - drhoPMP))) * c1o2;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //
      // Bernd das Brot 
	  //
      //
	  // x------x
	  // |      |
	  // |	 ---+--->X
	  // |		|  \
	  // x------x   \
	  //			off-vector
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      a0 = a0 + xoff * ax + yoff * ay + zoff * az + xoff_sq * axx + yoff_sq * ayy + zoff_sq * azz + xoff*yoff*axy + xoff*zoff*axz + yoff*zoff*ayz;
      ax = ax + c2o1 * xoff * axx + yoff * axy + zoff * axz;
      ay = ay + c2o1 * yoff * ayy + xoff * axy + zoff * ayz;
      az = az + c2o1 * zoff * azz + xoff * axz + yoff * ayz;
      b0 = b0 + xoff * bx + yoff * by + zoff * bz + xoff_sq * bxx + yoff_sq * byy + zoff_sq * bzz + xoff*yoff*bxy + xoff*zoff*bxz + yoff*zoff*byz;
      bx = bx + c2o1 * xoff * bxx + yoff * bxy + zoff * bxz;
      by = by + c2o1 * yoff * byy + xoff * bxy + zoff * byz;
      bz = bz + c2o1 * zoff * bzz + xoff * bxz + yoff * byz;
      c0 = c0 + xoff * cx + yoff * cy + zoff * cz + xoff_sq * cxx + yoff_sq * cyy + zoff_sq * czz + xoff*yoff*cxy + xoff*zoff*cxz + yoff*zoff*cyz;
      cx = cx + c2o1 * xoff * cxx + yoff * cxy + zoff * cxz;
      cy = cy + c2o1 * yoff * cyy + xoff * cxy + zoff * cyz;
      cz = cz + c2o1 * zoff * czz + xoff * cxz + yoff * cyz;
	  d0 = d0 + xoff * dx + yoff * dy + zoff * dz + xoff*yoff*dxy + xoff*zoff*dxz + yoff*zoff*dyz;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real mfcbb = c0;
	  real mfabb = c0;
	  real mfbcb = c0;
	  real mfbab = c0;
	  real mfbbc = c0;
	  real mfbba = c0;
	  real mfccb = c0;
	  real mfaab = c0;
	  real mfcab = c0;
	  real mfacb = c0;
	  real mfcbc = c0;
	  real mfaba = c0;
	  real mfcba = c0;
	  real mfabc = c0;
	  real mfbcc = c0;
	  real mfbaa = c0;
	  real mfbca = c0;
	  real mfbac = c0;
	  real mfbbb = c0;
	  real mfccc = c0;
	  real mfaac = c0;
	  real mfcac = c0;
	  real mfacc = c0;
	  real mfcca = c0;
	  real mfaaa = c0;
	  real mfcaa = c0;
	  real mfaca = c0;
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real mgcbb = c0;
	  real mgabb = c0;
	  real mgbcb = c0;
	  real mgbab = c0;
	  real mgbbc = c0;
	  real mgbba = c0;
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real m0, m1, m2, oMdrho;
	  real mxxPyyPzz, mxxMyy, mxxMzz, mxxyPyzz, mxxyMyzz, mxxzPyyz, mxxzMyyz, mxyyPxzz, mxyyMxzz;
	  //real qudricLimit = c1o100;//ganz schlechte Idee -> muss global sein
	  //real O3 = c2o1 - o;
	  //real residu, residutmp;
	  //residutmp = c0;///*-*/ c2o9 * (1./o - c1o2) * eps_new * eps_new;
	  real NeqOn = c1o1;//zero;//one;   //.... one = on ..... zero = off 
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //
	  //Position C 0., 0., 0.
	  //
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //x = 0.;
	  //y = 0.;
	  //z = 0.;
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real mxoff = -xoff;
	  //real myoff = -yoff;
	  //real mzoff = -zoff;
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //press = pressPPP * (c1o8 - c1o4 * mxoff - c1o4 * myoff - c1o4 * mzoff) + 
			//  pressMPP * (c1o8 + c1o4 * mxoff - c1o4 * myoff - c1o4 * mzoff) + 
			//  pressPMP * (c1o8 - c1o4 * mxoff + c1o4 * myoff - c1o4 * mzoff) + 
			//  pressMMP * (c1o8 + c1o4 * mxoff + c1o4 * myoff - c1o4 * mzoff) + 
			//  pressPPM * (c1o8 - c1o4 * mxoff - c1o4 * myoff + c1o4 * mzoff) + 
			//  pressMPM * (c1o8 + c1o4 * mxoff - c1o4 * myoff + c1o4 * mzoff) + 
			//  pressPMM * (c1o8 - c1o4 * mxoff + c1o4 * myoff + c1o4 * mzoff) + 
			//  pressMMM * (c1o8 + c1o4 * mxoff + c1o4 * myoff + c1o4 * mzoff);
	  //drho  = drhoPPP * (c1o8 - c1o4 * xoff - c1o4 * yoff - c1o4 * zoff) + 
			//  drhoMPP * (c1o8 + c1o4 * xoff - c1o4 * yoff - c1o4 * zoff) + 
			//  drhoPMP * (c1o8 - c1o4 * xoff + c1o4 * yoff - c1o4 * zoff) + 
			//  drhoMMP * (c1o8 + c1o4 * xoff + c1o4 * yoff - c1o4 * zoff) + 
			//  drhoPPM * (c1o8 - c1o4 * xoff - c1o4 * yoff + c1o4 * zoff) + 
			//  drhoMPM * (c1o8 + c1o4 * xoff - c1o4 * yoff + c1o4 * zoff) + 
			//  drhoPMM * (c1o8 - c1o4 * xoff + c1o4 * yoff + c1o4 * zoff) + 
			//  drhoMMM * (c1o8 + c1o4 * xoff + c1o4 * yoff + c1o4 * zoff);
	  press = d0;
	  vvx   = a0;
	  vvy   = b0;
	  vvz   = c0;

	  //mfaaa = drho;
	  //mfaaa = press + (ax+by+cz)/three;  //  1/3 = 2/3*(1/op-1/2)
	  mfaaa = press; // if drho is interpolated directly

	  vx2 = vvx*vvx;
	  vy2 = vvy*vvy;
	  vz2 = vvz*vvz;
	  oMdrho = c1o1;
	  //oMdrho = one - mfaaa;

	  //two
	  // linear combinations
	  real oP = o;// :(
	  mxxPyyPzz = mfaaa    -c2o3*(ax+by+cz)*eps_new/oP*(c1o1+press); 
	  //mxxMyy    = -c2o3*(ax - by)*eps_new/o;
	  //mxxMzz    = -c2o3*(ax - cz)*eps_new/o;

	  //mfabb     = -c1o3 * (bz + cy)*eps_new/o;
	  //mfbab     = -c1o3 * (az + cx)*eps_new/o;
	  //mfbba     = -c1o3 * (ay + bx)*eps_new/o;
	  mxxMyy    = -c2o3*((ax - by)+kxxMyyAverage)*eps_new/o * (c1o1 + press);
	  mxxMzz    = -c2o3*((ax - cz)+kxxMzzAverage)*eps_new/o * (c1o1 + press);

	  mfabb     = -c1o3 * ((bz + cy)+kyzAverage)*eps_new/o * (c1o1 + press);
	  mfbab     = -c1o3 * ((az + cx)+kxzAverage)*eps_new/o * (c1o1 + press);
	  mfbba     = -c1o3 * ((ay + bx)+kxyAverage)*eps_new/o * (c1o1 + press);

	  
	  // linear combinations back
	  mfcaa = c1o3 * (       mxxMyy +       mxxMzz + mxxPyyPzz) * NeqOn;
	  mfaca = c1o3 * (-c2o1 * mxxMyy +       mxxMzz + mxxPyyPzz) * NeqOn;
	  mfaac = c1o3 * (       mxxMyy - c2o1 * mxxMzz + mxxPyyPzz) * NeqOn;

	  //3.
	  // linear combinations
	  //residu = residutmp * (ayz + bxz + cxy );
	  //mfbbb = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mfbbb = c0;

	  //residu = residutmp * (axy + two*bxx + two*bzz + cyz );
	  //residu = -(c1o9*(axy - 2*bxx - 2*bzz + cyz ));
	  //mxxyPyzz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxxyPyzz = c0;

	  //residu = residutmp * (axy + two*bxx - two*bzz - cyz );
	  //residu = c1o9*(axy - 2*bxx + 2*bzz - cyz );
	  //mxxyMyzz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxxyMyzz = c0;

	  //residu = residutmp * (axz + byz + two*cxx + two*cyy );
	  //residu = -(c1o9*(axz + byz - 2*cxx - 2*cyy ));
	  //mxxzPyyz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxxzPyyz = c0;

	  //residu = residutmp * (axz - byz + two*cxx - two*cyy );
	  //residu = c1o9*(axz - byz - 2*cxx + 2*cyy );
	  //mxxzMyyz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxxzMyyz = c0;

	  //residu = residutmp * (two*ayy + two*azz + bxy + cxz );
	  //residu = c1o9*(2*ayy + 2*azz - bxy - cxz );
	  //mxyyPxzz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxyyPxzz = c0;

	  //residu = residutmp * (two*ayy - two*azz + bxy - cxz );
	  //residu = c1o9*(-2*ayy + 2*azz + bxy - cxz );
	  //mxyyMxzz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxyyMxzz = c0;

	  ////////////////////////////////////////////////////////////////////////////////////
	  // D3Q27F 
	  mgcbb =  (vvx * axx + ax * ax) * (eps_new * eps_new) * (c1o1 + press);
	  mgabb = -(vvx * axx + ax * ax) * (eps_new * eps_new) * (c1o1 + press);
	  mgbcb =  (vvy * byy + by * by) * (eps_new * eps_new) * (c1o1 + press);
	  mgbab = -(vvy * byy + by * by) * (eps_new * eps_new) * (c1o1 + press);
	  mgbbc =  (vvz * czz + cz * cz) * (eps_new * eps_new) * (c1o1 + press);
	  mgbba = -(vvz * czz + cz * cz) * (eps_new * eps_new) * (c1o1 + press);
	  //mgcbb = zero;
	  //mgabb = zero;
	  //mgbcb = zero;
	  //mgbab = zero;
	  //mgbbc = zero;
	  //mgbba = zero;
	  ////////////////////////////////////////////////////////////////////////////////////

	  // linear combinations back
	  mfcba = ( mxxyMyzz + mxxyPyzz) * c1o2;
	  mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
	  mfcab = ( mxxzMyyz + mxxzPyyz) * c1o2;
	  mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
	  mfbca = ( mxyyMxzz + mxyyPxzz) * c1o2;
	  mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;

	  //4.
	  mfacc = mfaaa*c1o9; 
	  mfcac = mfacc; 
	  mfcca = mfacc; 
	  //5.

	  //6.
	  mfccc = mfaaa*c1o27;
	  ////////////////////////////////////////////////////////////////////////////////////
	  //back
	  ////////////////////////////////////////////////////////////////////////////////////
	  //mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
	  ////////////////////////////////////////////////////////////////////////////////////
	  // Z - Dir
	  m0 =  mfaac * c1o2 +      mfaab * (vvz - c1o2) + (mfaaa + c1o1 * oMdrho) * (     vz2 - vvz) * c1o2; 
	  m1 = -mfaac        - c2o1 * mfaab *  vvz         +  mfaaa                * (c1o1 - vz2)              - c1o1 * oMdrho * vz2; 
	  m2 =  mfaac * c1o2 +      mfaab * (vvz + c1o2) + (mfaaa + c1o1 * oMdrho) * (     vz2 + vvz) * c1o2;
	  mfaaa = m0;
	  mfaab = m1;
	  mfaac = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfabc * c1o2 +      mfabb * (vvz - c1o2) + mfaba * (     vz2 - vvz) * c1o2; 
	  m1 = -mfabc        - c2o1 * mfabb *  vvz         + mfaba * (c1o1 - vz2); 
	  m2 =  mfabc * c1o2 +      mfabb * (vvz + c1o2) + mfaba * (     vz2 + vvz) * c1o2;
	  mfaba = m0;
	  mfabb = m1;
	  mfabc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfacc * c1o2 +      mfacb * (vvz - c1o2) + (mfaca + c1o3 * oMdrho) * (     vz2 - vvz) * c1o2; 
	  m1 = -mfacc        - c2o1 * mfacb *  vvz         +  mfaca                  * (c1o1 - vz2)              - c1o3 * oMdrho * vz2; 
	  m2 =  mfacc * c1o2 +      mfacb * (vvz + c1o2) + (mfaca + c1o3 * oMdrho) * (     vz2 + vvz) * c1o2;
	  mfaca = m0;
	  mfacb = m1;
	  mfacc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfbac * c1o2 +      mfbab * (vvz - c1o2) + mfbaa * (     vz2 - vvz) * c1o2; 
	  m1 = -mfbac        - c2o1 * mfbab *  vvz         + mfbaa * (c1o1 - vz2); 
	  m2 =  mfbac * c1o2 +      mfbab * (vvz + c1o2) + mfbaa * (     vz2 + vvz) * c1o2;
	  mfbaa = m0;
	  mfbab = m1;
	  mfbac = m2;
	  /////////b//////////////////////////////////////////////////////////////////////////
	  m0 =  mfbbc * c1o2 +      mfbbb * (vvz - c1o2) + mfbba * (     vz2 - vvz) * c1o2; 
	  m1 = -mfbbc        - c2o1 * mfbbb *  vvz         + mfbba * (c1o1 - vz2); 
	  m2 =  mfbbc * c1o2 +      mfbbb * (vvz + c1o2) + mfbba * (     vz2 + vvz) * c1o2;
	  mfbba = m0;
	  mfbbb = m1;
	  mfbbc = m2;
	  /////////b//////////////////////////////////////////////////////////////////////////
	  m0 =  mfbcc * c1o2 +      mfbcb * (vvz - c1o2) + mfbca * (     vz2 - vvz) * c1o2; 
	  m1 = -mfbcc        - c2o1 * mfbcb *  vvz         + mfbca * (c1o1 - vz2); 
	  m2 =  mfbcc * c1o2 +      mfbcb * (vvz + c1o2) + mfbca * (     vz2 + vvz) * c1o2;
	  mfbca = m0;
	  mfbcb = m1;
	  mfbcc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcac * c1o2 +      mfcab * (vvz - c1o2) + (mfcaa + c1o3 * oMdrho) * (     vz2 - vvz) * c1o2; 
	  m1 = -mfcac        - c2o1 * mfcab *  vvz         +  mfcaa                  * (c1o1 - vz2)              - c1o3 * oMdrho * vz2; 
	  m2 =  mfcac * c1o2 +      mfcab * (vvz + c1o2) + (mfcaa + c1o3 * oMdrho) * (     vz2 + vvz) * c1o2;
	  mfcaa = m0;
	  mfcab = m1;
	  mfcac = m2;
	  /////////c//////////////////////////////////////////////////////////////////////////
	  m0 =  mfcbc * c1o2 +      mfcbb * (vvz - c1o2) + mfcba * (     vz2 - vvz) * c1o2; 
	  m1 = -mfcbc        - c2o1 * mfcbb *  vvz         + mfcba * (c1o1 - vz2); 
	  m2 =  mfcbc * c1o2 +      mfcbb * (vvz + c1o2) + mfcba * (     vz2 + vvz) * c1o2;
	  mfcba = m0;
	  mfcbb = m1;
	  mfcbc = m2;
	  /////////c//////////////////////////////////////////////////////////////////////////
	  m0 =  mfccc * c1o2 +      mfccb * (vvz - c1o2) + (mfcca + c1o9 * oMdrho) * (     vz2 - vvz) * c1o2; 
	  m1 = -mfccc        - c2o1 * mfccb *  vvz         +  mfcca                  * (c1o1 - vz2)              - c1o9 * oMdrho * vz2; 
	  m2 =  mfccc * c1o2 +      mfccb * (vvz + c1o2) + (mfcca + c1o9 * oMdrho) * (     vz2 + vvz) * c1o2;
	  mfcca = m0;
	  mfccb = m1;
	  mfccc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  //mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
	  ////////////////////////////////////////////////////////////////////////////////////
	  // Y - Dir
	  m0 =  mfaca * c1o2 +      mfaba * (vvy - c1o2) + (mfaaa + c1o6 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfaca        - c2o1 * mfaba *  vvy         +  mfaaa                  * (c1o1 - vy2)              - c1o6 * oMdrho * vy2; 
	  m2 =  mfaca * c1o2 +      mfaba * (vvy + c1o2) + (mfaaa + c1o6 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfaaa = m0;
	  mfaba = m1;
	  mfaca = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfacb * c1o2 +      mfabb * (vvy - c1o2) + (mfaab + c2o3 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfacb        - c2o1 * mfabb *  vvy         +  mfaab                  * (c1o1 - vy2)              - c2o3 * oMdrho * vy2; 
	  m2 =  mfacb * c1o2 +      mfabb * (vvy + c1o2) + (mfaab + c2o3 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfaab = m0;
	  mfabb = m1;
	  mfacb = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfacc * c1o2 +      mfabc * (vvy - c1o2) + (mfaac + c1o6 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfacc        - c2o1 * mfabc *  vvy         +  mfaac                  * (c1o1 - vy2)              - c1o6 * oMdrho * vy2; 
	  m2 =  mfacc * c1o2 +      mfabc * (vvy + c1o2) + (mfaac + c1o6 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfaac = m0;
	  mfabc = m1;
	  mfacc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfbca * c1o2 +      mfbba * (vvy - c1o2) + mfbaa * (     vy2 - vvy) * c1o2; 
	  m1 = -mfbca        - c2o1 * mfbba *  vvy         + mfbaa * (c1o1 - vy2); 
	  m2 =  mfbca * c1o2 +      mfbba * (vvy + c1o2) + mfbaa * (     vy2 + vvy) * c1o2;
	  mfbaa = m0;
	  mfbba = m1;
	  mfbca = m2;
	  /////////b//////////////////////////////////////////////////////////////////////////
	  m0 =  mfbcb * c1o2 +      mfbbb * (vvy - c1o2) + mfbab * (     vy2 - vvy) * c1o2; 
	  m1 = -mfbcb        - c2o1 * mfbbb *  vvy         + mfbab * (c1o1 - vy2); 
	  m2 =  mfbcb * c1o2 +      mfbbb * (vvy + c1o2) + mfbab * (     vy2 + vvy) * c1o2;
	  mfbab = m0;
	  mfbbb = m1;
	  mfbcb = m2;
	  /////////b//////////////////////////////////////////////////////////////////////////
	  m0 =  mfbcc * c1o2 +      mfbbc * (vvy - c1o2) + mfbac * (     vy2 - vvy) * c1o2; 
	  m1 = -mfbcc        - c2o1 * mfbbc *  vvy         + mfbac * (c1o1 - vy2); 
	  m2 =  mfbcc * c1o2 +      mfbbc * (vvy + c1o2) + mfbac * (     vy2 + vvy) * c1o2;
	  mfbac = m0;
	  mfbbc = m1;
	  mfbcc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcca * c1o2 +      mfcba * (vvy - c1o2) + (mfcaa + c1o18 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfcca        - c2o1 * mfcba *  vvy         +  mfcaa                   * (c1o1 - vy2)              - c1o18 * oMdrho * vy2; 
	  m2 =  mfcca * c1o2 +      mfcba * (vvy + c1o2) + (mfcaa + c1o18 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfcaa = m0;
	  mfcba = m1;
	  mfcca = m2;
	  /////////c//////////////////////////////////////////////////////////////////////////
	  m0 =  mfccb * c1o2 +      mfcbb * (vvy - c1o2) + (mfcab + c2o9 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfccb        - c2o1 * mfcbb *  vvy         +  mfcab                  * (c1o1 - vy2)              - c2o9 * oMdrho * vy2; 
	  m2 =  mfccb * c1o2 +      mfcbb * (vvy + c1o2) + (mfcab + c2o9 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfcab = m0;
	  mfcbb = m1;
	  mfccb = m2;
	  /////////c//////////////////////////////////////////////////////////////////////////
	  m0 =  mfccc * c1o2 +      mfcbc * (vvy - c1o2) + (mfcac + c1o18 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfccc        - c2o1 * mfcbc *  vvy         +  mfcac                   * (c1o1 - vy2)              - c1o18 * oMdrho * vy2; 
	  m2 =  mfccc * c1o2 +      mfcbc * (vvy + c1o2) + (mfcac + c1o18 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfcac = m0;
	  mfcbc = m1;
	  mfccc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  //mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
	  ////////////////////////////////////////////////////////////////////////////////////
	  // X - Dir
	  m0 =  mfcaa * c1o2 +      mfbaa * (vvx - c1o2) + (mfaaa + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcaa        - c2o1 * mfbaa *  vvx         +  mfaaa                   * (c1o1 - vx2)              - c1o36 * oMdrho * vx2; 
	  m2 =  mfcaa * c1o2 +      mfbaa * (vvx + c1o2) + (mfaaa + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaaa = m0;
	  mfbaa = m1;
	  mfcaa = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcba * c1o2 +      mfbba * (vvx - c1o2) + (mfaba + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcba        - c2o1 * mfbba *  vvx         +  mfaba                  * (c1o1 - vx2)              - c1o9 * oMdrho * vx2; 
	  m2 =  mfcba * c1o2 +      mfbba * (vvx + c1o2) + (mfaba + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaba = m0;
	  mfbba = m1;
	  mfcba = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcca * c1o2 +      mfbca * (vvx - c1o2) + (mfaca + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcca        - c2o1 * mfbca *  vvx         +  mfaca                   * (c1o1 - vx2)              - c1o36 * oMdrho * vx2; 
	  m2 =  mfcca * c1o2 +      mfbca * (vvx + c1o2) + (mfaca + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaca = m0;
	  mfbca = m1;
	  mfcca = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcab * c1o2 +      mfbab * (vvx - c1o2) + (mfaab + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcab        - c2o1 * mfbab *  vvx         +  mfaab                  * (c1o1 - vx2)              - c1o9 * oMdrho * vx2; 
	  m2 =  mfcab * c1o2 +      mfbab * (vvx + c1o2) + (mfaab + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaab = m0;
	  mfbab = m1;
	  mfcab = m2;
	  ///////////b////////////////////////////////////////////////////////////////////////
	  m0 =  mfcbb * c1o2 +      mfbbb * (vvx - c1o2) + (mfabb + c4o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcbb        - c2o1 * mfbbb *  vvx         +  mfabb                  * (c1o1 - vx2)              - c4o9 * oMdrho * vx2; 
	  m2 =  mfcbb * c1o2 +      mfbbb * (vvx + c1o2) + (mfabb + c4o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfabb = m0;
	  mfbbb = m1;
	  mfcbb = m2;
	  ///////////b////////////////////////////////////////////////////////////////////////
	  m0 =  mfccb * c1o2 +      mfbcb * (vvx - c1o2) + (mfacb + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfccb        - c2o1 * mfbcb *  vvx         +  mfacb                  * (c1o1 - vx2)              - c1o9 * oMdrho * vx2; 
	  m2 =  mfccb * c1o2 +      mfbcb * (vvx + c1o2) + (mfacb + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfacb = m0;
	  mfbcb = m1;
	  mfccb = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcac * c1o2 +      mfbac * (vvx - c1o2) + (mfaac + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcac        - c2o1 * mfbac *  vvx         +  mfaac                   * (c1o1 - vx2)              - c1o36 * oMdrho * vx2; 
	  m2 =  mfcac * c1o2 +      mfbac * (vvx + c1o2) + (mfaac + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaac = m0;
	  mfbac = m1;
	  mfcac = m2;
	  ///////////c////////////////////////////////////////////////////////////////////////
	  m0 =  mfcbc * c1o2 +      mfbbc * (vvx - c1o2) + (mfabc + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcbc        - c2o1 * mfbbc *  vvx         +  mfabc                  * (c1o1 - vx2)              - c1o9 * oMdrho * vx2; 
	  m2 =  mfcbc * c1o2 +      mfbbc * (vvx + c1o2) + (mfabc + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfabc = m0;
	  mfbbc = m1;
	  mfcbc = m2;
	  ///////////c////////////////////////////////////////////////////////////////////////
	  m0 =  mfccc * c1o2 +      mfbcc * (vvx - c1o2) + (mfacc + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfccc        - c2o1 * mfbcc *  vvx         +  mfacc                   * (c1o1 - vx2)              - c1o36 * oMdrho * vx2; 
	  m2 =  mfccc * c1o2 +      mfbcc * (vvx + c1o2) + (mfacc + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfacc = m0;
	  mfbcc = m1;
	  mfccc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////

	  ////////////////////////////////////////////////////////////////////////////////////
	  //index 0
	  k000 = posC[k];
	  kM00 = neighborCX[k000];
	  k0M0 = neighborCY[k000];
	  k00M = neighborCZ[k000];
	  kMM0 = neighborCY[kM00];
	  kM0M = neighborCZ[kM00];
	  k0MM = neighborCZ[k0M0];
	  kMMM = neighborCZ[kMM0];
	  ////////////////////////////////////////////////////////////////////////////////////

	  ////////////////////////////////////////////////////////////////////////////////////
	  (G.g[DIR_P00])[k000] = mgcbb;
	  (G.g[DIR_M00])[kM00] = mgabb;
	  (G.g[DIR_0P0])[k000] = mgbcb;
	  (G.g[DIR_0M0])[k0M0] = mgbab;
	  (G.g[DIR_00P])[k000] = mgbbc;
	  (G.g[DIR_00M])[k00M] = mgbba;
	  ////////////////////////////////////////////////////////////////////////////////////
	  fP00dest[k000] = mfcbb;                                                                 
	  fM00dest[kM00] = mfabb;                                                               
	  f0P0dest[k000] = mfbcb;
	  f0M0dest[k0M0] = mfbab;
	  f00Pdest[k000] = mfbbc;
	  f00Mdest[k00M] = mfbba;
	  fPP0dest[k000] = mfccb;
	  fMM0dest[kMM0] = mfaab;
	  fPM0dest[k0M0] = mfcab;
	  fMP0dest[kM00] = mfacb;
	  fP0Pdest[k000] = mfcbc;
	  fM0Mdest[kM0M] = mfaba;
	  fP0Mdest[k00M] = mfcba;
	  fM0Pdest[kM00] = mfabc;
	  f0PPdest[k000] = mfbcc;
	  f0MMdest[k0MM] = mfbaa;
	  f0PMdest[k00M] = mfbca;
	  f0MPdest[k0M0] = mfbac;
	  f000dest[k000] = mfbbb;
	  fMMMdest[kMMM] = mfaaa;
	  fMMPdest[kMM0] = mfaac;
	  fMPPdest[kM00] = mfacc;
	  fMPMdest[kM0M] = mfaca;
	  fPPMdest[k00M] = mfcca;
	  fPPPdest[k000] = mfccc;
	  fPMPdest[k0M0] = mfcac;
	  fPMMdest[k0MM] = mfcaa;
	  ////////////////////////////////////////////////////////////////////////////////////
   }
}
//////////////////////////////////////////////////////////////////////////






















































//////////////////////////////////////////////////////////////////////////
__global__ void scaleFC_comp_D3Q27F3( real* DC,
												 real* DF,
												 real* G6,
												 unsigned int* neighborCX,
												 unsigned int* neighborCY,
												 unsigned int* neighborCZ,
												 unsigned int* neighborFX,
												 unsigned int* neighborFY,
												 unsigned int* neighborFZ,
												 unsigned long long numberOfLBnodesCoarse, 
												 unsigned long long numberOfLBnodesFine, 
												 bool isEvenTimestep,
												 unsigned int* posC, 
												 unsigned int* posFSWB, 
												 unsigned int kFC, 
												 real omCoarse, 
												 real omFine, 
												 real nu, 
												 unsigned int nxC, 
												 unsigned int nyC, 
												 unsigned int nxF, 
												 unsigned int nyF,
												 ICellNeighFC offFC)
{
   real 
	   *fP00source, *fM00source, *f0P0source, *f0M0source, *f00Psource, *f00Msource, *fPP0source, *fMM0source, *fPM0source,
	   *fMP0source, *fP0Psource, *fM0Msource, *fP0Msource, *fM0Psource, *f0PPsource, *f0MMsource, *f0PMsource, *f0MPsource,
	   *f000source, *fMMMsource, *fMMPsource, *fMPPsource, *fMPMsource, *fPPMsource, *fPPPsource, *fPMPsource, *fPMMsource;


   fP00source = &DF[DIR_P00 * numberOfLBnodesFine];
   fM00source = &DF[DIR_M00 * numberOfLBnodesFine];
   f0P0source = &DF[DIR_0P0 * numberOfLBnodesFine];
   f0M0source = &DF[DIR_0M0 * numberOfLBnodesFine];
   f00Psource = &DF[DIR_00P * numberOfLBnodesFine];
   f00Msource = &DF[DIR_00M * numberOfLBnodesFine];
   fPP0source = &DF[DIR_PP0 * numberOfLBnodesFine];
   fMM0source = &DF[DIR_MM0 * numberOfLBnodesFine];
   fPM0source = &DF[DIR_PM0 * numberOfLBnodesFine];
   fMP0source = &DF[DIR_MP0 * numberOfLBnodesFine];
   fP0Psource = &DF[DIR_P0P * numberOfLBnodesFine];
   fM0Msource = &DF[DIR_M0M * numberOfLBnodesFine];
   fP0Msource = &DF[DIR_P0M * numberOfLBnodesFine];
   fM0Psource = &DF[DIR_M0P * numberOfLBnodesFine];
   f0PPsource = &DF[DIR_0PP * numberOfLBnodesFine];
   f0MMsource = &DF[DIR_0MM * numberOfLBnodesFine];
   f0PMsource = &DF[DIR_0PM * numberOfLBnodesFine];
   f0MPsource = &DF[DIR_0MP * numberOfLBnodesFine];
   f000source = &DF[DIR_000 * numberOfLBnodesFine];
   fMMMsource = &DF[DIR_MMM * numberOfLBnodesFine];
   fMMPsource = &DF[DIR_MMP * numberOfLBnodesFine];
   fMPPsource = &DF[DIR_MPP * numberOfLBnodesFine];
   fMPMsource = &DF[DIR_MPM * numberOfLBnodesFine];
   fPPMsource = &DF[DIR_PPM * numberOfLBnodesFine];
   fPPPsource = &DF[DIR_PPP * numberOfLBnodesFine];
   fPMPsource = &DF[DIR_PMP * numberOfLBnodesFine];
   fPMMsource = &DF[DIR_PMM * numberOfLBnodesFine];

   real
	   *fP00dest, *fM00dest, *f0P0dest, *f0M0dest, *f00Pdest, *f00Mdest, *fPP0dest, *fMM0dest, *fPM0dest,
	   *fMP0dest, *fP0Pdest, *fM0Mdest, *fP0Mdest, *fM0Pdest, *f0PPdest, *f0MMdest, *f0PMdest, *f0MPdest,
	   *f000dest, *fMMMdest, *fMMPdest, *fMPPdest, *fMPMdest, *fPPMdest, *fPPPdest, *fPMPdest, *fPMMdest;

   if (isEvenTimestep==true)
   {
	   fP00dest = &DC[DIR_P00 * numberOfLBnodesCoarse];
	   fM00dest = &DC[DIR_M00 * numberOfLBnodesCoarse];
	   f0P0dest = &DC[DIR_0P0 * numberOfLBnodesCoarse];
	   f0M0dest = &DC[DIR_0M0 * numberOfLBnodesCoarse];
	   f00Pdest = &DC[DIR_00P * numberOfLBnodesCoarse];
	   f00Mdest = &DC[DIR_00M * numberOfLBnodesCoarse];
	   fPP0dest = &DC[DIR_PP0 * numberOfLBnodesCoarse];
	   fMM0dest = &DC[DIR_MM0 * numberOfLBnodesCoarse];
	   fPM0dest = &DC[DIR_PM0 * numberOfLBnodesCoarse];
	   fMP0dest = &DC[DIR_MP0 * numberOfLBnodesCoarse];
	   fP0Pdest = &DC[DIR_P0P * numberOfLBnodesCoarse];
	   fM0Mdest = &DC[DIR_M0M * numberOfLBnodesCoarse];
	   fP0Mdest = &DC[DIR_P0M * numberOfLBnodesCoarse];
	   fM0Pdest = &DC[DIR_M0P * numberOfLBnodesCoarse];
	   f0PPdest = &DC[DIR_0PP * numberOfLBnodesCoarse];
	   f0MMdest = &DC[DIR_0MM * numberOfLBnodesCoarse];
	   f0PMdest = &DC[DIR_0PM * numberOfLBnodesCoarse];
	   f0MPdest = &DC[DIR_0MP * numberOfLBnodesCoarse];
	   f000dest = &DC[DIR_000 * numberOfLBnodesCoarse];
	   fMMMdest = &DC[DIR_MMM * numberOfLBnodesCoarse];
	   fMMPdest = &DC[DIR_MMP * numberOfLBnodesCoarse];
	   fMPPdest = &DC[DIR_MPP * numberOfLBnodesCoarse];
	   fMPMdest = &DC[DIR_MPM * numberOfLBnodesCoarse];
	   fPPMdest = &DC[DIR_PPM * numberOfLBnodesCoarse];
	   fPPPdest = &DC[DIR_PPP * numberOfLBnodesCoarse];
	   fPMPdest = &DC[DIR_PMP * numberOfLBnodesCoarse];
	   fPMMdest = &DC[DIR_PMM * numberOfLBnodesCoarse];
   } 
   else
   {
	   fP00dest = &DC[DIR_M00 * numberOfLBnodesCoarse];
	   fM00dest = &DC[DIR_P00 * numberOfLBnodesCoarse];
	   f0P0dest = &DC[DIR_0M0 * numberOfLBnodesCoarse];
	   f0M0dest = &DC[DIR_0P0 * numberOfLBnodesCoarse];
	   f00Pdest = &DC[DIR_00M * numberOfLBnodesCoarse];
	   f00Mdest = &DC[DIR_00P * numberOfLBnodesCoarse];
	   fPP0dest = &DC[DIR_MM0 * numberOfLBnodesCoarse];
	   fMM0dest = &DC[DIR_PP0 * numberOfLBnodesCoarse];
	   fPM0dest = &DC[DIR_MP0 * numberOfLBnodesCoarse];
	   fMP0dest = &DC[DIR_PM0 * numberOfLBnodesCoarse];
	   fP0Pdest = &DC[DIR_M0M * numberOfLBnodesCoarse];
	   fM0Mdest = &DC[DIR_P0P * numberOfLBnodesCoarse];
	   fP0Mdest = &DC[DIR_M0P * numberOfLBnodesCoarse];
	   fM0Pdest = &DC[DIR_P0M * numberOfLBnodesCoarse];
	   f0PPdest = &DC[DIR_0MM * numberOfLBnodesCoarse];
	   f0MMdest = &DC[DIR_0PP * numberOfLBnodesCoarse];
	   f0PMdest = &DC[DIR_0MP * numberOfLBnodesCoarse];
	   f0MPdest = &DC[DIR_0PM * numberOfLBnodesCoarse];
	   f000dest = &DC[DIR_000 * numberOfLBnodesCoarse];
	   fMMMdest = &DC[DIR_PPP * numberOfLBnodesCoarse];
	   fMMPdest = &DC[DIR_PPM * numberOfLBnodesCoarse];
	   fMPPdest = &DC[DIR_PMM * numberOfLBnodesCoarse];
	   fMPMdest = &DC[DIR_PMP * numberOfLBnodesCoarse];
	   fPPMdest = &DC[DIR_MMP * numberOfLBnodesCoarse];
	   fPPPdest = &DC[DIR_MMM * numberOfLBnodesCoarse];
	   fPMPdest = &DC[DIR_MPM * numberOfLBnodesCoarse];
	   fPMMdest = &DC[DIR_MPP * numberOfLBnodesCoarse];
   }

   Distributions6 G;
   if (isEvenTimestep == true)
   {
	   G.g[DIR_P00] = &G6[DIR_P00 * numberOfLBnodesCoarse];
	   G.g[DIR_M00] = &G6[DIR_M00 * numberOfLBnodesCoarse];
	   G.g[DIR_0P0] = &G6[DIR_0P0 * numberOfLBnodesCoarse];
	   G.g[DIR_0M0] = &G6[DIR_0M0 * numberOfLBnodesCoarse];
	   G.g[DIR_00P] = &G6[DIR_00P * numberOfLBnodesCoarse];
	   G.g[DIR_00M] = &G6[DIR_00M * numberOfLBnodesCoarse];
   }
   else
   {
	   G.g[DIR_M00] = &G6[DIR_P00 * numberOfLBnodesCoarse];
	   G.g[DIR_P00] = &G6[DIR_M00 * numberOfLBnodesCoarse];
	   G.g[DIR_0M0] = &G6[DIR_0P0 * numberOfLBnodesCoarse];
	   G.g[DIR_0P0] = &G6[DIR_0M0 * numberOfLBnodesCoarse];
	   G.g[DIR_00M] = &G6[DIR_00P * numberOfLBnodesCoarse];
	   G.g[DIR_00P] = &G6[DIR_00M * numberOfLBnodesCoarse];
   }

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  ix = threadIdx.x;  // Globaler x-Index 
   const unsigned  iy = blockIdx.x;   // Globaler y-Index 
   const unsigned  iz = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*iz + iy) + ix;
   //////////////////////////////////////////////////////////////////////////

   ////////////////////////////////////////////////////////////////////////////////
   real eps_new = c2o1;
   real omegaS = omFine;//-omFine;
   real o  = omCoarse;//-omCoarse;
   //real op = one;
   //real cu_sq;

   real xoff,    yoff,    zoff;
   real xoff_sq, yoff_sq, zoff_sq;

   // real drho;
   real        vvx, vvy, vvz, vx2, vy2, vz2;
   real        press;//,drho,vx1,vx2,vx3;
   real        /*pressMMM,*/drhoMMM,vx1MMM,vx2MMM,vx3MMM;
   real        /*pressMMP,*/drhoMMP,vx1MMP,vx2MMP,vx3MMP;
   real        /*pressMPP,*/drhoMPP,vx1MPP,vx2MPP,vx3MPP;
   real        /*pressMPM,*/drhoMPM,vx1MPM,vx2MPM,vx3MPM;
   real        /*pressPPM,*/drhoPPM,vx1PPM,vx2PPM,vx3PPM;
   real        /*pressPPP,*/drhoPPP,vx1PPP,vx2PPP,vx3PPP;
   real        /*pressPMP,*/drhoPMP,vx1PMP,vx2PMP,vx3PMP;
   real        /*pressPMM,*/drhoPMM,vx1PMM,vx2PMM,vx3PMM;
   real        fP00,fM00,f0P0,f0M0,f00P,f00M,fPP0,fMM0,fPM0,fMP0,fP0P,fM0M,fP0M,fM0P,f0PP,f0MM,f0PM,f0MP,f000,fPPP, fMMP, fPMP, fMPP, fPPM, fMMM, fPMM, fMPM;
   real        kxyFromfcNEQMMM, kyzFromfcNEQMMM, kxzFromfcNEQMMM, kxxMyyFromfcNEQMMM, kxxMzzFromfcNEQMMM, kyyMzzFromfcNEQMMM;
   real        kxyFromfcNEQMMP, kyzFromfcNEQMMP, kxzFromfcNEQMMP, kxxMyyFromfcNEQMMP, kxxMzzFromfcNEQMMP, kyyMzzFromfcNEQMMP;
   real        kxyFromfcNEQMPP, kyzFromfcNEQMPP, kxzFromfcNEQMPP, kxxMyyFromfcNEQMPP, kxxMzzFromfcNEQMPP, kyyMzzFromfcNEQMPP;
   real        kxyFromfcNEQMPM, kyzFromfcNEQMPM, kxzFromfcNEQMPM, kxxMyyFromfcNEQMPM, kxxMzzFromfcNEQMPM, kyyMzzFromfcNEQMPM;
   real        kxyFromfcNEQPPM, kyzFromfcNEQPPM, kxzFromfcNEQPPM, kxxMyyFromfcNEQPPM, kxxMzzFromfcNEQPPM, kyyMzzFromfcNEQPPM;
   real        kxyFromfcNEQPPP, kyzFromfcNEQPPP, kxzFromfcNEQPPP, kxxMyyFromfcNEQPPP, kxxMzzFromfcNEQPPP, kyyMzzFromfcNEQPPP;
   real        kxyFromfcNEQPMP, kyzFromfcNEQPMP, kxzFromfcNEQPMP, kxxMyyFromfcNEQPMP, kxxMzzFromfcNEQPMP, kyyMzzFromfcNEQPMP;
   real        kxyFromfcNEQPMM, kyzFromfcNEQPMM, kxzFromfcNEQPMM, kxxMyyFromfcNEQPMM, kxxMzzFromfcNEQPMM, kyyMzzFromfcNEQPMM;
   real        a0, ax, ay, az, axx, ayy, azz, axy, axz, ayz, b0, bx, by, bz, bxx, byy, bzz, bxy, bxz, byz, c0, cx, cy, cz, cxx, cyy, czz, cxy, cxz, cyz/*, axyz, bxyz, cxyz*/;
   real        d0, dx, dy, dz, dxy, dxz, dyz/*, dxyz*/;

   if(k<kFC)
   {
      //////////////////////////////////////////////////////////////////////////
      xoff = offFC.x[k];
      yoff = offFC.y[k];
      zoff = offFC.z[k];      
      xoff_sq = xoff * xoff;
      yoff_sq = yoff * yoff;
      zoff_sq = zoff * zoff;
      //////////////////////////////////////////////////////////////////////////
      //SWB//
      //////////////////////////////////////////////////////////////////////////
      //index 0
      unsigned int k000base = posFSWB[k];
      unsigned int kM00base = neighborFX[k000base];
      unsigned int k0M0base = neighborFY[k000base];
      unsigned int k00Mbase = neighborFZ[k000base];
      unsigned int kMM0base = neighborFY[kM00base];
      unsigned int kM0Mbase = neighborFZ[kM00base];
      unsigned int k0MMbase = neighborFZ[k0M0base];
      unsigned int kMMMbase = neighborFZ[kMM0base];
      //////////////////////////////////////////////////////////////////////////
      //index 
      unsigned int k000 = k000base;
      unsigned int kM00 = kM00base;   
      unsigned int k0M0 = k0M0base;   
      unsigned int k00M = k00Mbase;   
      unsigned int kMM0 = kMM0base;  
      unsigned int kM0M = kM0Mbase;  
      unsigned int k0MM = k0MMbase;  
      unsigned int kMMM = kMMMbase; 
      ////////////////////////////////////////////////////////////////////////////////
      fP00 = fP00source[k000];
      fM00 = fM00source[kM00];
      f0P0 = f0P0source[k000];
      f0M0 = f0M0source[k0M0];
      f00P = f00Psource[k000];
      f00M = f00Msource[k00M];
      fPP0 = fPP0source[k000];
      fMM0 = fMM0source[kMM0];
      fPM0 = fPM0source[k0M0];
      fMP0 = fMP0source[kM00];
      fP0P = fP0Psource[k000];
      fM0M = fM0Msource[kM0M];
      fP0M = fP0Msource[k00M];
      fM0P = fM0Psource[kM00];
      f0PP = f0PPsource[k000];
      f0MM = f0MMsource[k0MM];
      f0PM = f0PMsource[k00M];
      f0MP = f0MPsource[k0M0];
      f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
      fPMP = fPMPsource[k0M0];
      fPMM = fPMMsource[k0MM];

      drhoMMM = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1MMM  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoMMM);
	  vx2MMM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoMMM);
	  vx3MMM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoMMM);

	  kxyFromfcNEQMMM    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoMMM) - ((vx1MMM*vx2MMM)));
	  kyzFromfcNEQMMM    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoMMM) - ((vx2MMM*vx3MMM)));
	  kxzFromfcNEQMMM    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoMMM) - ((vx1MMM*vx3MMM)));
	  kxxMyyFromfcNEQMMM = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoMMM) - ((vx1MMM*vx1MMM - vx2MMM*vx2MMM)));
	  kxxMzzFromfcNEQMMM = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoMMM) - ((vx1MMM*vx1MMM - vx3MMM*vx3MMM)));
	  kyyMzzFromfcNEQMMM = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoMMM) - ((vx2MMM*vx2MMM - vx3MMM*vx3MMM)));

      //////////////////////////////////////////////////////////////////////////
      //SWT//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = k00M;
      kM00 = kM0M;   
      k0M0 = k0MM;   
      k00M = neighborFZ[k00M];   
      kMM0 = kMMM;  
      kM0M = neighborFZ[kM0M];  
      k0MM = neighborFZ[k0MM];  
      kMMM = neighborFZ[kMMM]; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoMMP = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1MMP  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoMMP);
	  vx2MMP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoMMP);
	  vx3MMP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoMMP);

	  kxyFromfcNEQMMP    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoMMP) - ((vx1MMP*vx2MMP)));
	  kyzFromfcNEQMMP    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoMMP) - ((vx2MMP*vx3MMP)));
	  kxzFromfcNEQMMP    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoMMP) - ((vx1MMP*vx3MMP)));
	  kxxMyyFromfcNEQMMP = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoMMP) - ((vx1MMP*vx1MMP - vx2MMP*vx2MMP)));
	  kxxMzzFromfcNEQMMP = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoMMP) - ((vx1MMP*vx1MMP - vx3MMP*vx3MMP)));
	  kyyMzzFromfcNEQMMP = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoMMP) - ((vx2MMP*vx2MMP - vx3MMP*vx3MMP)));

      //////////////////////////////////////////////////////////////////////////
      //SET//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = kM00;
      kM00 = neighborFX[kM00];   
      k0M0 = kMM0;   
      k00M = kM0M;   
      kMM0 = neighborFX[kMM0];  
      kM0M = neighborFX[kM0M];  
      k0MM = kMMM;  
      kMMM = neighborFX[kMMM]; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoPMP = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1PMP  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoPMP);
	  vx2PMP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoPMP);
	  vx3PMP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoPMP);

	  kxyFromfcNEQPMP    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoPMP) - ((vx1PMP*vx2PMP)));
	  kyzFromfcNEQPMP    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoPMP) - ((vx2PMP*vx3PMP)));
	  kxzFromfcNEQPMP    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoPMP) - ((vx1PMP*vx3PMP)));
	  kxxMyyFromfcNEQPMP = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoPMP) - ((vx1PMP*vx1PMP - vx2PMP*vx2PMP)));
	  kxxMzzFromfcNEQPMP = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoPMP) - ((vx1PMP*vx1PMP - vx3PMP*vx3PMP)));
	  kyyMzzFromfcNEQPMP = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoPMP) - ((vx2PMP*vx2PMP - vx3PMP*vx3PMP)));

      //////////////////////////////////////////////////////////////////////////
      //SEB//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k00M = k000;   
      kM0M = kM00;  
      k0MM = k0M0;  
      kMMM = kMM0; 
      k000 = kM00base;
      kM00 = neighborFX[kM00base];   
      k0M0 = kMM0base;   
      kMM0 = neighborFX[kMM0base];  
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoPMM = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1PMM  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoPMM);
	  vx2PMM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoPMM);
	  vx3PMM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoPMM);

	  kxyFromfcNEQPMM    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoPMM) - ((vx1PMM*vx2PMM)));
	  kyzFromfcNEQPMM    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoPMM) - ((vx2PMM*vx3PMM)));
	  kxzFromfcNEQPMM    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoPMM) - ((vx1PMM*vx3PMM)));
	  kxxMyyFromfcNEQPMM = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoPMM) - ((vx1PMM*vx1PMM - vx2PMM*vx2PMM)));
	  kxxMzzFromfcNEQPMM = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoPMM) - ((vx1PMM*vx1PMM - vx3PMM*vx3PMM)));
	  kyyMzzFromfcNEQPMM = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoPMM) - ((vx2PMM*vx2PMM - vx3PMM*vx3PMM)));

      //////////////////////////////////////////////////////////////////////////
      //NWB//
      //////////////////////////////////////////////////////////////////////////
      //index 0
      k000base = k0M0base;
      kM00base = kMM0base;
      k0M0base = neighborFY[k0M0base];
      k00Mbase = k0MMbase;
      kMM0base = neighborFY[kMM0base];
      kM0Mbase = kMMMbase;
      k0MMbase = neighborFY[k0MMbase];
      kMMMbase = neighborFY[kMMMbase];
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = k000base;
      kM00 = kM00base;   
      k0M0 = k0M0base;   
      k00M = k00Mbase;   
      kMM0 = kMM0base;  
      kM0M = kM0Mbase;  
      k0MM = k0MMbase;  
      kMMM = kMMMbase; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoMPM = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1MPM  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoMPM);
	  vx2MPM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoMPM);
	  vx3MPM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoMPM);

	  kxyFromfcNEQMPM    = -c3o1*omegaS*   ((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoMPM) - ((vx1MPM*vx2MPM)));
	  kyzFromfcNEQMPM    = -c3o1*omegaS*   ((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoMPM) - ((vx2MPM*vx3MPM)));
	  kxzFromfcNEQMPM    = -c3o1*omegaS*   ((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoMPM) - ((vx1MPM*vx3MPM)));
	  kxxMyyFromfcNEQMPM = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoMPM) - ((vx1MPM*vx1MPM - vx2MPM*vx2MPM)));
	  kxxMzzFromfcNEQMPM = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoMPM) - ((vx1MPM*vx1MPM - vx3MPM*vx3MPM)));
	  kyyMzzFromfcNEQMPM = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoMPM) - ((vx2MPM*vx2MPM - vx3MPM*vx3MPM)));

	  //////////////////////////////////////////////////////////////////////////
      //NWT//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = k00M;
      kM00 = kM0M;   
      k0M0 = k0MM;   
      k00M = neighborFZ[k00M];   
      kMM0 = kMMM;  
      kM0M = neighborFZ[kM0M];  
      k0MM = neighborFZ[k0MM];  
      kMMM = neighborFZ[kMMM]; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoMPP = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1MPP  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoMPP);
	  vx2MPP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoMPP);
	  vx3MPP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoMPP);

	  kxyFromfcNEQMPP    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoMPP) - ((vx1MPP*vx2MPP)));
	  kyzFromfcNEQMPP    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoMPP) - ((vx2MPP*vx3MPP)));
	  kxzFromfcNEQMPP    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoMPP) - ((vx1MPP*vx3MPP)));
	  kxxMyyFromfcNEQMPP = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoMPP) - ((vx1MPP*vx1MPP - vx2MPP*vx2MPP)));
	  kxxMzzFromfcNEQMPP = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoMPP) - ((vx1MPP*vx1MPP - vx3MPP*vx3MPP)));
	  kyyMzzFromfcNEQMPP = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoMPP) - ((vx2MPP*vx2MPP - vx3MPP*vx3MPP)));

      //////////////////////////////////////////////////////////////////////////
      //NET//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k000 = kM00;
      kM00 = neighborFX[kM00];   
      k0M0 = kMM0;   
      k00M = kM0M;   
      kMM0 = neighborFX[kMM0];  
      kM0M = neighborFX[kM0M];  
      k0MM = kMMM;  
      kMMM = neighborFX[kMMM]; 
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoPPP = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1PPP  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoPPP);
	  vx2PPP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoPPP);
	  vx3PPP  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoPPP);

	  kxyFromfcNEQPPP    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoPPP) - ((vx1PPP*vx2PPP)));
	  kyzFromfcNEQPPP    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoPPP) - ((vx2PPP*vx3PPP)));
	  kxzFromfcNEQPPP    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoPPP) - ((vx1PPP*vx3PPP)));
	  kxxMyyFromfcNEQPPP = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoPPP) - ((vx1PPP*vx1PPP - vx2PPP*vx2PPP)));
	  kxxMzzFromfcNEQPPP = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoPPP) - ((vx1PPP*vx1PPP - vx3PPP*vx3PPP)));
	  kyyMzzFromfcNEQPPP = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoPPP) - ((vx2PPP*vx2PPP - vx3PPP*vx3PPP)));

      //////////////////////////////////////////////////////////////////////////
      //NEB//
      //////////////////////////////////////////////////////////////////////////
      //index 
      k00M = k000;   
      kM0M = kM00;  
      k0MM = k0M0;  
      kMMM = kMM0; 
      k000 = kM00base;
      kM00 = neighborFX[kM00base];   
      k0M0 = kMM0base;   
      kMM0 = neighborFX[kMM0base];  
      ////////////////////////////////////////////////////////////////////////////////
	  fP00 = fP00source[k000];
	  fM00 = fM00source[kM00];
	  f0P0 = f0P0source[k000];
	  f0M0 = f0M0source[k0M0];
	  f00P = f00Psource[k000];
	  f00M = f00Msource[k00M];
	  fPP0 = fPP0source[k000];
	  fMM0 = fMM0source[kMM0];
	  fPM0 = fPM0source[k0M0];
	  fMP0 = fMP0source[kM00];
	  fP0P = fP0Psource[k000];
	  fM0M = fM0Msource[kM0M];
	  fP0M = fP0Msource[k00M];
	  fM0P = fM0Psource[kM00];
	  f0PP = f0PPsource[k000];
	  f0MM = f0MMsource[k0MM];
	  f0PM = f0PMsource[k00M];
	  f0MP = f0MPsource[k0M0];
	  f000 = f000source[k000];
	  fMMM = fMMMsource[kMMM];
	  fMMP = fMMPsource[kMM0];
	  fMPP = fMPPsource[kM00];
	  fMPM = fMPMsource[kM0M];
	  fPPM = fPPMsource[k00M];
	  fPPP = fPPPsource[k000];
	  fPMP = fPMPsource[k0M0];
	  fPMM = fPMMsource[k0MM];

      drhoPPM = fP00+fM00+f0P0+f0M0+f00P+f00M+fPP0+fMM0+fPM0+fMP0+fP0P+fM0M+fP0M+fM0P+f0PP+f0MM+f0PM+f0MP+f000+fPPP+fMMP+fPMP+fMPP+fPPM+fMMM+fPMM+fMPM;
      vx1PPM  = (((fPPP-fMMM)+(fPMP-fMPM)+(fPPM-fMMP)+(fPMM-fMPP)) + (((fPP0-fMM0)+(fP0P-fM0M))+((fPM0-fMP0)+(fP0M-fM0P))) + (fP00-fM00))/(c1o1 + drhoPPM);
	  vx2PPM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPPM-fMMP)+(fMPM-fPMP)) + (((fPP0-fMM0)+(f0PP-f0MM))+((f0PM-f0MP)+(fMP0-fPM0))) + (f0P0-f0M0))/(c1o1 + drhoPPM);
	  vx3PPM  = (((fPPP-fMMM)+(fMPP-fPMM)+(fPMP-fMPM)+(fMMP-fPPM)) + (((fP0P-fM0M)+(f0PP-f0MM))+((fM0P-fP0M)+(f0MP-f0PM))) + (f00P-f00M))/(c1o1 + drhoPPM);

	  kxyFromfcNEQPPM    = -c3o1*omegaS*((((fMM0 - fPM0) + (fPP0 - fMP0)) + (((fMMM - fPMM) + (fPPM - fMPM)) + ((fMMP - fPMP) + (fPPP - fMPP)))) / (c1o1 + drhoPPM) - ((vx1PPM*vx2PPM)));
	  kyzFromfcNEQPPM    = -c3o1*omegaS*((((f0MM - f0MP) + (f0PP - f0PM)) + (((fPMM - fPMP) + (fMMM - fMPM)) + ((fPPP - fPPM) + (fMPP - fMMP)))) / (c1o1 + drhoPPM) - ((vx2PPM*vx3PPM)));
	  kxzFromfcNEQPPM    = -c3o1*omegaS*((((fM0M - fP0M) + (fP0P - fM0P)) + (((fMMM - fPMM) + (fMPM - fPPM)) + ((fPMP - fMMP) + (fPPP - fMPP)))) / (c1o1 + drhoPPM) - ((vx1PPM*vx3PPM)));
	  kxxMyyFromfcNEQPPM = -c3o2*omegaS *(((((fM0M - f0MM) + (fM0P - f0MP)) + ((fP0M - f0PM) + (fP0P - f0PP))) + ((fM00 - f0M0) + (fP00 - f0P0))) / (c1o1 + drhoPPM) - ((vx1PPM*vx1PPM - vx2PPM*vx2PPM)));
	  kxxMzzFromfcNEQPPM = -c3o2*omegaS *(((((fMM0 - f0MM) + (fMP0 - f0PM)) + ((fPM0 - f0MP) + (fPP0 - f0PP))) + ((fM00 - f00M) + (fP00 - f00P))) / (c1o1 + drhoPPM) - ((vx1PPM*vx1PPM - vx3PPM*vx3PPM)));
	  kyyMzzFromfcNEQPPM = -c3o2*omegaS *(((((fPM0 - fP0M) + (fMM0 - fM0M)) + ((fPP0 - fP0P) + (fMP0 - fM0P))) + ((f0M0 - f00M) + (f0P0 - f00P))) / (c1o1 + drhoPPM) - ((vx2PPM*vx2PPM - vx3PPM*vx3PPM)));

      //////////////////////////////////////////////////////////////////////////
      //3
      //////////////////////////////////////////////////////////////////////////
	  a0  = c1o8*(((vx1PPM + vx1MMP) + (vx1MPM + vx1PMP)) + ((vx1PMM + vx1MPP) + (vx1MMM + vx1PPP)));
	  ax  = c1o4*(((vx1PPM - vx1MMP) + (vx1PPP - vx1MMM)) + ((vx1PMP - vx1MPM) + (vx1PMM - vx1MPP)));
	  ay  = c1o4*(((vx1PPM - vx1MMP) + (vx1PPP - vx1MMM)) + ((vx1MPM - vx1PMP) + (vx1MPP - vx1PMM)));
	  az  = c1o4*(((vx1MMP - vx1PPM) + (vx1PPP - vx1MMM)) + ((vx1PMP - vx1MPM) + (vx1MPP - vx1PMM)));
	  axy = c1o2*(((vx1PPM - vx1PMP) + (vx1MMM - vx1MPP)) + ((vx1MMP - vx1MPM) + (vx1PPP - vx1PMM)));
	  axz = c1o2*(((vx1PMP - vx1PPM) + (vx1MMM - vx1MPP)) + ((vx1MPM - vx1MMP) + (vx1PPP - vx1PMM)));
	  ayz = c1o2*(((vx1PPP - vx1MPM) + (vx1PMM - vx1MMP)) + ((vx1MPP - vx1PPM) + (vx1MMM - vx1PMP)));
	  //axyz=		  ((vx1MMP - vx1PPM) + (vx1PPP - vx1MMM)) + ((vx1MPM - vx1PMP) + (vx1PMM - vx1MPP)) ;
	  b0  = c1o8*(((vx2PPM + vx2MMP) + (vx2MPM + vx2PMP)) + ((vx2PMM + vx2MPP) + (vx2MMM + vx2PPP)));
	  bx  = c1o4*(((vx2PPM - vx2MMP) + (vx2PPP - vx2MMM)) + ((vx2PMP - vx2MPM) + (vx2PMM - vx2MPP)));
	  by  = c1o4*(((vx2PPM - vx2MMP) + (vx2PPP - vx2MMM)) + ((vx2MPM - vx2PMP) + (vx2MPP - vx2PMM)));
	  bz  = c1o4*(((vx2MMP - vx2PPM) + (vx2PPP - vx2MMM)) + ((vx2PMP - vx2MPM) + (vx2MPP - vx2PMM)));
	  bxy = c1o2*(((vx2PPM - vx2MPP) + (vx2MMM - vx2PMP)) + ((vx2MMP - vx2PMM) + (vx2PPP - vx2MPM)));
	  bxz = c1o2*(((vx2MMM - vx2PPM) + (vx2PMP - vx2MPP)) + ((vx2MPM - vx2PMM) + (vx2PPP - vx2MMP)));
	  byz = c1o2*(((vx2MPP - vx2PPM) + (vx2MMM - vx2PMP)) + ((vx2PMM - vx2MMP) + (vx2PPP - vx2MPM)));
	  //bxyz=		  ((vx2MMP - vx2PPM) + (vx2PPP - vx2MMM)) + ((vx2MPM - vx2PMP) + (vx2PMM - vx2MPP)) ;
	  c0  = c1o8*(((vx3PPM + vx3MMP) + (vx3MPM + vx3PMP)) + ((vx3PMM + vx3MPP) + (vx3MMM + vx3PPP)));
	  cx  = c1o4*(((vx3PPM - vx3MMP) + (vx3PPP - vx3MMM)) + ((vx3PMP - vx3MPM) + (vx3PMM - vx3MPP)));
	  cy  = c1o4*(((vx3PPM - vx3MMP) + (vx3PPP - vx3MMM)) + ((vx3MPM - vx3PMP) + (vx3MPP - vx3PMM)));
	  cz  = c1o4*(((vx3MMP - vx3PPM) + (vx3PPP - vx3MMM)) + ((vx3PMP - vx3MPM) + (vx3MPP - vx3PMM)));
	  cxy = c1o2*(((vx3PPM - vx3PMP) + (vx3MMM - vx3MPP)) + ((vx3MMP - vx3MPM) + (vx3PPP - vx3PMM)));
	  cxz = c1o2*(((vx3MMM - vx3PPM) + (vx3PMP - vx3MPP)) + ((vx3MPM - vx3PMM) + (vx3PPP - vx3MMP)));
	  cyz = c1o2*(((vx3MMM - vx3PPM) + (vx3MPP - vx3PMP)) + ((vx3PMM - vx3MPM) + (vx3PPP - vx3MMP)));
	  //cxyz=		  ((vx3MMP - vx3PPM) + (vx3PPP - vx3MMM)) + ((vx3MPM - vx3PMP) + (vx3PMM - vx3MPP)) ;

	  //bxx = zero;
	  //cxx = zero;
	  //ayy = zero;
	  //cyy = zero;
	  //azz = zero;
	  //bzz = zero;
	  //axx = zero;
	  //byy = zero;
	  //czz = zero;

	  bxx = c1o8*(((kxyFromfcNEQPPP - kxyFromfcNEQMMM) + (kxyFromfcNEQPMM - kxyFromfcNEQMPP)) + ((kxyFromfcNEQPMP - kxyFromfcNEQMPM) + (kxyFromfcNEQPPM - kxyFromfcNEQMMP))) - c1o2*axy;
	  cxx = c1o8*(((kxzFromfcNEQPPP - kxzFromfcNEQMMM) + (kxzFromfcNEQPMM - kxzFromfcNEQMPP)) + ((kxzFromfcNEQPMP - kxzFromfcNEQMPM) + (kxzFromfcNEQPPM - kxzFromfcNEQMMP))) - c1o2*axz;

	  ayy = c1o8*(((kxyFromfcNEQPPP - kxyFromfcNEQMMM) + (kxyFromfcNEQPPM - kxyFromfcNEQMMP)) - ((kxyFromfcNEQPMM - kxyFromfcNEQMPP) + (kxyFromfcNEQPMP - kxyFromfcNEQMPM))) - c1o2*bxy;
	  cyy = c1o8*(((kyzFromfcNEQPPP - kyzFromfcNEQMMM) + (kyzFromfcNEQPPM - kyzFromfcNEQMMP)) - ((kyzFromfcNEQPMM - kyzFromfcNEQMPP) + (kyzFromfcNEQPMP - kyzFromfcNEQMPM))) - c1o2*byz;

	  azz = c1o8*(((kxzFromfcNEQPPP - kxzFromfcNEQMMM) - (kxzFromfcNEQPMM - kxzFromfcNEQMPP)) + ((kxzFromfcNEQPMP - kxzFromfcNEQMPM) - (kxzFromfcNEQPPM - kxzFromfcNEQMMP))) - c1o2*cxz;
	  bzz = c1o8*(((kyzFromfcNEQPPP - kyzFromfcNEQMMM) - (kyzFromfcNEQPMM - kyzFromfcNEQMPP)) + ((kyzFromfcNEQPMP - kyzFromfcNEQMPM) - (kyzFromfcNEQPPM - kyzFromfcNEQMMP))) - c1o2*cyz;

	  axx = ( c1o16*(((kxxMyyFromfcNEQPPP - kxxMyyFromfcNEQMMM) + (kxxMyyFromfcNEQPMM - kxxMyyFromfcNEQMPP)) + ((kxxMyyFromfcNEQPMP - kxxMyyFromfcNEQMPM) + (kxxMyyFromfcNEQPPM - kxxMyyFromfcNEQMMP))) - c1o4*bxy)
		  + ( c1o16*(((kxxMzzFromfcNEQPPP - kxxMzzFromfcNEQMMM) + (kxxMzzFromfcNEQPMM - kxxMzzFromfcNEQMPP)) + ((kxxMzzFromfcNEQPMP - kxxMzzFromfcNEQMPM) + (kxxMzzFromfcNEQPPM - kxxMzzFromfcNEQMMP))) - c1o4*cxz);

	  byy = (-c1o16*(((kxxMyyFromfcNEQPPP - kxxMyyFromfcNEQMMM) - (kxxMyyFromfcNEQPMM - kxxMyyFromfcNEQMPP)) + ((kxxMyyFromfcNEQPPM - kxxMyyFromfcNEQMMP) - (kxxMyyFromfcNEQPMP - kxxMyyFromfcNEQMPM))) - c1o4*axy)
		  + ( c1o16*(((kyyMzzFromfcNEQPPP - kyyMzzFromfcNEQMMM) - (kyyMzzFromfcNEQPMM - kyyMzzFromfcNEQMPP)) + ((kyyMzzFromfcNEQPPM - kyyMzzFromfcNEQMMP) - (kyyMzzFromfcNEQPMP - kyyMzzFromfcNEQMPM))) - c1o4*cyz);

	  czz = (-c1o16*(((kxxMzzFromfcNEQPPP - kxxMzzFromfcNEQMMM) - (kxxMzzFromfcNEQPMM - kxxMzzFromfcNEQMPP)) - ((kxxMzzFromfcNEQPPM - kxxMzzFromfcNEQMMP) - (kxxMzzFromfcNEQPMP - kxxMzzFromfcNEQMPM))) - c1o4*axz)
		  + ( c1o16*(((kyyMzzFromfcNEQPPP - kyyMzzFromfcNEQMMM) - (kyyMzzFromfcNEQPMM - kyyMzzFromfcNEQMPP)) - ((kyyMzzFromfcNEQPPM - kyyMzzFromfcNEQMMP) - (kyyMzzFromfcNEQPMP - kyyMzzFromfcNEQMPM))) - c1o4*byz);

	  a0 -= c1o4*(axx + ayy + azz);
	  b0 -= c1o4*(bxx + byy + bzz);
	  c0 -= c1o4*(cxx + cyy + czz);

	  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real kxyAverage = c0;
	  real kyzAverage = c0;
	  real kxzAverage = c0;
	  real kxxMyyAverage = c0;
	  real kxxMzzAverage = c0;
	  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////Press
	  //d0   = ( pressPPM + pressPPP + pressMPM + pressMPP + pressPMM + pressPMP + pressMMM + pressMMP) * c1o8;
	  //dx   = ( pressPPM + pressPPP - pressMPM - pressMPP + pressPMM + pressPMP - pressMMM - pressMMP) * c1o4;
	  //dy   = ( pressPPM + pressPPP + pressMPM + pressMPP - pressPMM - pressPMP - pressMMM - pressMMP) * c1o4;
	  //dz   = (-pressPPM + pressPPP - pressMPM + pressMPP - pressPMM + pressPMP - pressMMM + pressMMP) * c1o4;
	  //dxy  = ( pressPPM + pressPPP - pressMPM - pressMPP - pressPMM - pressPMP + pressMMM + pressMMP) * c1o2;
	  //dxz  = (-pressPPM + pressPPP + pressMPM - pressMPP - pressPMM + pressPMP + pressMMM - pressMMP) * c1o2;
	  //dyz  = (-pressPPM + pressPPP - pressMPM + pressMPP + pressPMM - pressPMP + pressMMM - pressMMP) * c1o2;
	  //dxyz =  -pressPPM + pressPPP + pressMPM - pressMPP + pressPMM - pressPMP - pressMMM + pressMMP;
	  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //drho
	  d0   = ( ((drhoPPM + drhoMMP) + (drhoMPM + drhoPMP)) + ((drhoPMM + drhoMPP) + (drhoMMM + drhoPPP))) * c1o8;
	  dx   = ( ((drhoPPM - drhoMMP) + (drhoPMM - drhoMPP)) + ((drhoPMP - drhoMPM) + (drhoPPP - drhoMMM))) * c1o4;
	  dy   = ( ((drhoPPM - drhoMMP) + (drhoMPP - drhoPMM)) + ((drhoMPM - drhoPMP) + (drhoPPP - drhoMMM))) * c1o4;
	  dz   = ( ((drhoMMP - drhoPPM) + (drhoPPP - drhoMMM)) + ((drhoPMP - drhoMPM) + (drhoMPP - drhoPMM))) * c1o4;
	  dxy  = ( ((drhoPPM - drhoPMP) + (drhoPPP - drhoPMM)) + ((drhoMMP - drhoMPM) + (drhoMMM - drhoMPP))) * c1o2;
	  dxz  = ( ((drhoMMM - drhoPPM) + (drhoPPP - drhoMMP)) + ((drhoMPM - drhoPMM) + (drhoPMP - drhoMPP))) * c1o2;
	  dyz  = ( ((drhoMPP - drhoPPM) + (drhoPPP - drhoMPM)) + ((drhoPMM - drhoMMP) + (drhoMMM - drhoPMP))) * c1o2;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //
      // Bernd das Brot 
	  //
      //
	  // x------x
	  // |      |
	  // |	 ---+--->X
	  // |		|  \
	  // x------x   \
	  //			off-vector
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      a0 = a0 + xoff * ax + yoff * ay + zoff * az + xoff_sq * axx + yoff_sq * ayy + zoff_sq * azz + xoff*yoff*axy + xoff*zoff*axz + yoff*zoff*ayz;
      ax = ax + c2o1 * xoff * axx + yoff * axy + zoff * axz;
      ay = ay + c2o1 * yoff * ayy + xoff * axy + zoff * ayz;
      az = az + c2o1 * zoff * azz + xoff * axz + yoff * ayz;
      b0 = b0 + xoff * bx + yoff * by + zoff * bz + xoff_sq * bxx + yoff_sq * byy + zoff_sq * bzz + xoff*yoff*bxy + xoff*zoff*bxz + yoff*zoff*byz;
      bx = bx + c2o1 * xoff * bxx + yoff * bxy + zoff * bxz;
      by = by + c2o1 * yoff * byy + xoff * bxy + zoff * byz;
      bz = bz + c2o1 * zoff * bzz + xoff * bxz + yoff * byz;
      c0 = c0 + xoff * cx + yoff * cy + zoff * cz + xoff_sq * cxx + yoff_sq * cyy + zoff_sq * czz + xoff*yoff*cxy + xoff*zoff*cxz + yoff*zoff*cyz;
      cx = cx + c2o1 * xoff * cxx + yoff * cxy + zoff * cxz;
      cy = cy + c2o1 * yoff * cyy + xoff * cxy + zoff * cyz;
      cz = cz + c2o1 * zoff * czz + xoff * cxz + yoff * cyz;
	  d0 = d0 + xoff * dx + yoff * dy + zoff * dz + xoff*yoff*dxy + xoff*zoff*dxz + yoff*zoff*dyz;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real mfcbb = c0;
	  real mfabb = c0;
	  real mfbcb = c0;
	  real mfbab = c0;
	  real mfbbc = c0;
	  real mfbba = c0;
	  real mfccb = c0;
	  real mfaab = c0;
	  real mfcab = c0;
	  real mfacb = c0;
	  real mfcbc = c0;
	  real mfaba = c0;
	  real mfcba = c0;
	  real mfabc = c0;
	  real mfbcc = c0;
	  real mfbaa = c0;
	  real mfbca = c0;
	  real mfbac = c0;
	  real mfbbb = c0;
	  real mfccc = c0;
	  real mfaac = c0;
	  real mfcac = c0;
	  real mfacc = c0;
	  real mfcca = c0;
	  real mfaaa = c0;
	  real mfcaa = c0;
	  real mfaca = c0;
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real mgcbb = c0;
	  real mgabb = c0;
	  real mgbcb = c0;
	  real mgbab = c0;
	  real mgbbc = c0;
	  real mgbba = c0;
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real m0, m1, m2, oMdrho;
	  real mxxPyyPzz, mxxMyy, mxxMzz, mxxyPyzz, mxxyMyzz, mxxzPyyz, mxxzMyyz, mxyyPxzz, mxyyMxzz;
	  //real qudricLimit = c1o100;//ganz schlechte Idee -> muss global sein
	  //real O3 = c2o1 - o;
	  //real residu, residutmp;
	  //residutmp = c0;///*-*/ c2o9 * (1./o - c1o2) * eps_new * eps_new;
	  real NeqOn = c1o1;//zero;//one;   //.... one = on ..... zero = off 
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //
	  //Position C 0., 0., 0.
	  //
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //x = 0.;
	  //y = 0.;
	  //z = 0.;
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real mxoff = -xoff;
	  //real myoff = -yoff;
	  //real mzoff = -zoff;
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //press = pressPPP * (c1o8 - c1o4 * mxoff - c1o4 * myoff - c1o4 * mzoff) + 
			//  pressMPP * (c1o8 + c1o4 * mxoff - c1o4 * myoff - c1o4 * mzoff) + 
			//  pressPMP * (c1o8 - c1o4 * mxoff + c1o4 * myoff - c1o4 * mzoff) + 
			//  pressMMP * (c1o8 + c1o4 * mxoff + c1o4 * myoff - c1o4 * mzoff) + 
			//  pressPPM * (c1o8 - c1o4 * mxoff - c1o4 * myoff + c1o4 * mzoff) + 
			//  pressMPM * (c1o8 + c1o4 * mxoff - c1o4 * myoff + c1o4 * mzoff) + 
			//  pressPMM * (c1o8 - c1o4 * mxoff + c1o4 * myoff + c1o4 * mzoff) + 
			//  pressMMM * (c1o8 + c1o4 * mxoff + c1o4 * myoff + c1o4 * mzoff);
	  //drho  = drhoPPP * (c1o8 - c1o4 * xoff - c1o4 * yoff - c1o4 * zoff) + 
			//  drhoMPP * (c1o8 + c1o4 * xoff - c1o4 * yoff - c1o4 * zoff) + 
			//  drhoPMP * (c1o8 - c1o4 * xoff + c1o4 * yoff - c1o4 * zoff) + 
			//  drhoMMP * (c1o8 + c1o4 * xoff + c1o4 * yoff - c1o4 * zoff) + 
			//  drhoPPM * (c1o8 - c1o4 * xoff - c1o4 * yoff + c1o4 * zoff) + 
			//  drhoMPM * (c1o8 + c1o4 * xoff - c1o4 * yoff + c1o4 * zoff) + 
			//  drhoPMM * (c1o8 - c1o4 * xoff + c1o4 * yoff + c1o4 * zoff) + 
			//  drhoMMM * (c1o8 + c1o4 * xoff + c1o4 * yoff + c1o4 * zoff);
	  press = d0;
	  vvx   = a0;
	  vvy   = b0;
	  vvz   = c0;

	  //mfaaa = drho;
	  //mfaaa = press + (ax+by+cz)/three;  //  1/3 = 2/3*(1/op-1/2)
	  mfaaa = press; // if drho is interpolated directly

	  vx2 = vvx*vvx;
	  vy2 = vvy*vvy;
	  vz2 = vvz*vvz;
	  oMdrho = c1o1;
	  //oMdrho = one - mfaaa;

	  //two
	  // linear combinations
	  real oP = o;// :(
	  mxxPyyPzz = mfaaa    -c2o3*(ax+by+cz)*eps_new/oP*(c1o1+press); 
	  //mxxMyy    = -c2o3*(ax - by)*eps_new/o;
	  //mxxMzz    = -c2o3*(ax - cz)*eps_new/o;

	  //mfabb     = -c1o3 * (bz + cy)*eps_new/o;
	  //mfbab     = -c1o3 * (az + cx)*eps_new/o;
	  //mfbba     = -c1o3 * (ay + bx)*eps_new/o;
	  mxxMyy    = -c2o3*((ax - by)+kxxMyyAverage)*eps_new/o * (c1o1 + press);
	  mxxMzz    = -c2o3*((ax - cz)+kxxMzzAverage)*eps_new/o * (c1o1 + press);

	  mfabb     = -c1o3 * ((bz + cy)+kyzAverage)*eps_new/o * (c1o1 + press);
	  mfbab     = -c1o3 * ((az + cx)+kxzAverage)*eps_new/o * (c1o1 + press);
	  mfbba     = -c1o3 * ((ay + bx)+kxyAverage)*eps_new/o * (c1o1 + press);

	  
	  // linear combinations back
	  mfcaa = c1o3 * (       mxxMyy +       mxxMzz + mxxPyyPzz) * NeqOn;
	  mfaca = c1o3 * (-c2o1 * mxxMyy +       mxxMzz + mxxPyyPzz) * NeqOn;
	  mfaac = c1o3 * (       mxxMyy - c2o1 * mxxMzz + mxxPyyPzz) * NeqOn;

	  //3.
	  // linear combinations
	  //residu = residutmp * (ayz + bxz + cxy );
	  //mfbbb = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mfbbb = c0;

	  //residu = residutmp * (axy + two*bxx + two*bzz + cyz );
	  //residu = -(c1o9*(axy - 2*bxx - 2*bzz + cyz ));
	  //mxxyPyzz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxxyPyzz = c0;

	  //residu = residutmp * (axy + two*bxx - two*bzz - cyz );
	  //residu = c1o9*(axy - 2*bxx + 2*bzz - cyz );
	  //mxxyMyzz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxxyMyzz = c0;

	  //residu = residutmp * (axz + byz + two*cxx + two*cyy );
	  //residu = -(c1o9*(axz + byz - 2*cxx - 2*cyy ));
	  //mxxzPyyz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxxzPyyz = c0;

	  //residu = residutmp * (axz - byz + two*cxx - two*cyy );
	  //residu = c1o9*(axz - byz - 2*cxx + 2*cyy );
	  //mxxzMyyz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxxzMyyz = c0;

	  //residu = residutmp * (two*ayy + two*azz + bxy + cxz );
	  //residu = c1o9*(2*ayy + 2*azz - bxy - cxz );
	  //mxyyPxzz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxyyPxzz = c0;

	  //residu = residutmp * (two*ayy - two*azz + bxy - cxz );
	  //residu = c1o9*(-2*ayy + 2*azz + bxy - cxz );
	  //mxyyMxzz = (abs(residu)+qudricLimit) * residu / (qudricLimit * O3 + abs(residu));
	  mxyyMxzz = c0;

	  ////////////////////////////////////////////////////////////////////////////////////
	  // D3Q27F 
	  mgcbb = (ax - c4o1 * axx) * eps_new;
	  mgabb = (ax + c4o1 * axx) * eps_new;
	  mgbcb = (by - c4o1 * byy) * eps_new;
	  mgbab = (by + c4o1 * byy) * eps_new;
	  mgbbc = (cz - c4o1 * czz) * eps_new;
	  mgbba = (cz + c4o1 * czz) * eps_new;
	  ////////////////////////////////////////////////////////////////////////////////////

	  // linear combinations back
	  mfcba = ( mxxyMyzz + mxxyPyzz) * c1o2;
	  mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
	  mfcab = ( mxxzMyyz + mxxzPyyz) * c1o2;
	  mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
	  mfbca = ( mxyyMxzz + mxyyPxzz) * c1o2;
	  mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;

	  //4.
	  mfacc = mfaaa*c1o9; 
	  mfcac = mfacc; 
	  mfcca = mfacc; 
	  //5.

	  //6.
	  mfccc = mfaaa*c1o27;
	  ////////////////////////////////////////////////////////////////////////////////////
	  //back
	  ////////////////////////////////////////////////////////////////////////////////////
	  //mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
	  ////////////////////////////////////////////////////////////////////////////////////
	  // Z - Dir
	  m0 =  mfaac * c1o2 +      mfaab * (vvz - c1o2) + (mfaaa + c1o1 * oMdrho) * (     vz2 - vvz) * c1o2; 
	  m1 = -mfaac        - c2o1 * mfaab *  vvz         +  mfaaa                * (c1o1 - vz2)              - c1o1 * oMdrho * vz2; 
	  m2 =  mfaac * c1o2 +      mfaab * (vvz + c1o2) + (mfaaa + c1o1 * oMdrho) * (     vz2 + vvz) * c1o2;
	  mfaaa = m0;
	  mfaab = m1;
	  mfaac = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfabc * c1o2 +      mfabb * (vvz - c1o2) + mfaba * (     vz2 - vvz) * c1o2; 
	  m1 = -mfabc        - c2o1 * mfabb *  vvz         + mfaba * (c1o1 - vz2); 
	  m2 =  mfabc * c1o2 +      mfabb * (vvz + c1o2) + mfaba * (     vz2 + vvz) * c1o2;
	  mfaba = m0;
	  mfabb = m1;
	  mfabc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfacc * c1o2 +      mfacb * (vvz - c1o2) + (mfaca + c1o3 * oMdrho) * (     vz2 - vvz) * c1o2; 
	  m1 = -mfacc        - c2o1 * mfacb *  vvz         +  mfaca                  * (c1o1 - vz2)              - c1o3 * oMdrho * vz2; 
	  m2 =  mfacc * c1o2 +      mfacb * (vvz + c1o2) + (mfaca + c1o3 * oMdrho) * (     vz2 + vvz) * c1o2;
	  mfaca = m0;
	  mfacb = m1;
	  mfacc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfbac * c1o2 +      mfbab * (vvz - c1o2) + mfbaa * (     vz2 - vvz) * c1o2; 
	  m1 = -mfbac        - c2o1 * mfbab *  vvz         + mfbaa * (c1o1 - vz2); 
	  m2 =  mfbac * c1o2 +      mfbab * (vvz + c1o2) + mfbaa * (     vz2 + vvz) * c1o2;
	  mfbaa = m0;
	  mfbab = m1;
	  mfbac = m2;
	  /////////b//////////////////////////////////////////////////////////////////////////
	  m0 =  mfbbc * c1o2 +      mfbbb * (vvz - c1o2) + mfbba * (     vz2 - vvz) * c1o2; 
	  m1 = -mfbbc        - c2o1 * mfbbb *  vvz         + mfbba * (c1o1 - vz2); 
	  m2 =  mfbbc * c1o2 +      mfbbb * (vvz + c1o2) + mfbba * (     vz2 + vvz) * c1o2;
	  mfbba = m0;
	  mfbbb = m1;
	  mfbbc = m2;
	  /////////b//////////////////////////////////////////////////////////////////////////
	  m0 =  mfbcc * c1o2 +      mfbcb * (vvz - c1o2) + mfbca * (     vz2 - vvz) * c1o2; 
	  m1 = -mfbcc        - c2o1 * mfbcb *  vvz         + mfbca * (c1o1 - vz2); 
	  m2 =  mfbcc * c1o2 +      mfbcb * (vvz + c1o2) + mfbca * (     vz2 + vvz) * c1o2;
	  mfbca = m0;
	  mfbcb = m1;
	  mfbcc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcac * c1o2 +      mfcab * (vvz - c1o2) + (mfcaa + c1o3 * oMdrho) * (     vz2 - vvz) * c1o2; 
	  m1 = -mfcac        - c2o1 * mfcab *  vvz         +  mfcaa                  * (c1o1 - vz2)              - c1o3 * oMdrho * vz2; 
	  m2 =  mfcac * c1o2 +      mfcab * (vvz + c1o2) + (mfcaa + c1o3 * oMdrho) * (     vz2 + vvz) * c1o2;
	  mfcaa = m0;
	  mfcab = m1;
	  mfcac = m2;
	  /////////c//////////////////////////////////////////////////////////////////////////
	  m0 =  mfcbc * c1o2 +      mfcbb * (vvz - c1o2) + mfcba * (     vz2 - vvz) * c1o2; 
	  m1 = -mfcbc        - c2o1 * mfcbb *  vvz         + mfcba * (c1o1 - vz2); 
	  m2 =  mfcbc * c1o2 +      mfcbb * (vvz + c1o2) + mfcba * (     vz2 + vvz) * c1o2;
	  mfcba = m0;
	  mfcbb = m1;
	  mfcbc = m2;
	  /////////c//////////////////////////////////////////////////////////////////////////
	  m0 =  mfccc * c1o2 +      mfccb * (vvz - c1o2) + (mfcca + c1o9 * oMdrho) * (     vz2 - vvz) * c1o2; 
	  m1 = -mfccc        - c2o1 * mfccb *  vvz         +  mfcca                  * (c1o1 - vz2)              - c1o9 * oMdrho * vz2; 
	  m2 =  mfccc * c1o2 +      mfccb * (vvz + c1o2) + (mfcca + c1o9 * oMdrho) * (     vz2 + vvz) * c1o2;
	  mfcca = m0;
	  mfccb = m1;
	  mfccc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  //mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
	  ////////////////////////////////////////////////////////////////////////////////////
	  // Y - Dir
	  m0 =  mfaca * c1o2 +      mfaba * (vvy - c1o2) + (mfaaa + c1o6 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfaca        - c2o1 * mfaba *  vvy         +  mfaaa                  * (c1o1 - vy2)              - c1o6 * oMdrho * vy2; 
	  m2 =  mfaca * c1o2 +      mfaba * (vvy + c1o2) + (mfaaa + c1o6 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfaaa = m0;
	  mfaba = m1;
	  mfaca = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfacb * c1o2 +      mfabb * (vvy - c1o2) + (mfaab + c2o3 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfacb        - c2o1 * mfabb *  vvy         +  mfaab                  * (c1o1 - vy2)              - c2o3 * oMdrho * vy2; 
	  m2 =  mfacb * c1o2 +      mfabb * (vvy + c1o2) + (mfaab + c2o3 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfaab = m0;
	  mfabb = m1;
	  mfacb = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfacc * c1o2 +      mfabc * (vvy - c1o2) + (mfaac + c1o6 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfacc        - c2o1 * mfabc *  vvy         +  mfaac                  * (c1o1 - vy2)              - c1o6 * oMdrho * vy2; 
	  m2 =  mfacc * c1o2 +      mfabc * (vvy + c1o2) + (mfaac + c1o6 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfaac = m0;
	  mfabc = m1;
	  mfacc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfbca * c1o2 +      mfbba * (vvy - c1o2) + mfbaa * (     vy2 - vvy) * c1o2; 
	  m1 = -mfbca        - c2o1 * mfbba *  vvy         + mfbaa * (c1o1 - vy2); 
	  m2 =  mfbca * c1o2 +      mfbba * (vvy + c1o2) + mfbaa * (     vy2 + vvy) * c1o2;
	  mfbaa = m0;
	  mfbba = m1;
	  mfbca = m2;
	  /////////b//////////////////////////////////////////////////////////////////////////
	  m0 =  mfbcb * c1o2 +      mfbbb * (vvy - c1o2) + mfbab * (     vy2 - vvy) * c1o2; 
	  m1 = -mfbcb        - c2o1 * mfbbb *  vvy         + mfbab * (c1o1 - vy2); 
	  m2 =  mfbcb * c1o2 +      mfbbb * (vvy + c1o2) + mfbab * (     vy2 + vvy) * c1o2;
	  mfbab = m0;
	  mfbbb = m1;
	  mfbcb = m2;
	  /////////b//////////////////////////////////////////////////////////////////////////
	  m0 =  mfbcc * c1o2 +      mfbbc * (vvy - c1o2) + mfbac * (     vy2 - vvy) * c1o2; 
	  m1 = -mfbcc        - c2o1 * mfbbc *  vvy         + mfbac * (c1o1 - vy2); 
	  m2 =  mfbcc * c1o2 +      mfbbc * (vvy + c1o2) + mfbac * (     vy2 + vvy) * c1o2;
	  mfbac = m0;
	  mfbbc = m1;
	  mfbcc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcca * c1o2 +      mfcba * (vvy - c1o2) + (mfcaa + c1o18 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfcca        - c2o1 * mfcba *  vvy         +  mfcaa                   * (c1o1 - vy2)              - c1o18 * oMdrho * vy2; 
	  m2 =  mfcca * c1o2 +      mfcba * (vvy + c1o2) + (mfcaa + c1o18 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfcaa = m0;
	  mfcba = m1;
	  mfcca = m2;
	  /////////c//////////////////////////////////////////////////////////////////////////
	  m0 =  mfccb * c1o2 +      mfcbb * (vvy - c1o2) + (mfcab + c2o9 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfccb        - c2o1 * mfcbb *  vvy         +  mfcab                  * (c1o1 - vy2)              - c2o9 * oMdrho * vy2; 
	  m2 =  mfccb * c1o2 +      mfcbb * (vvy + c1o2) + (mfcab + c2o9 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfcab = m0;
	  mfcbb = m1;
	  mfccb = m2;
	  /////////c//////////////////////////////////////////////////////////////////////////
	  m0 =  mfccc * c1o2 +      mfcbc * (vvy - c1o2) + (mfcac + c1o18 * oMdrho) * (     vy2 - vvy) * c1o2; 
	  m1 = -mfccc        - c2o1 * mfcbc *  vvy         +  mfcac                   * (c1o1 - vy2)              - c1o18 * oMdrho * vy2; 
	  m2 =  mfccc * c1o2 +      mfcbc * (vvy + c1o2) + (mfcac + c1o18 * oMdrho) * (     vy2 + vvy) * c1o2;
	  mfcac = m0;
	  mfcbc = m1;
	  mfccc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  //mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
	  ////////////////////////////////////////////////////////////////////////////////////
	  // X - Dir
	  m0 =  mfcaa * c1o2 +      mfbaa * (vvx - c1o2) + (mfaaa + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcaa        - c2o1 * mfbaa *  vvx         +  mfaaa                   * (c1o1 - vx2)              - c1o36 * oMdrho * vx2; 
	  m2 =  mfcaa * c1o2 +      mfbaa * (vvx + c1o2) + (mfaaa + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaaa = m0;
	  mfbaa = m1;
	  mfcaa = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcba * c1o2 +      mfbba * (vvx - c1o2) + (mfaba + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcba        - c2o1 * mfbba *  vvx         +  mfaba                  * (c1o1 - vx2)              - c1o9 * oMdrho * vx2; 
	  m2 =  mfcba * c1o2 +      mfbba * (vvx + c1o2) + (mfaba + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaba = m0;
	  mfbba = m1;
	  mfcba = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcca * c1o2 +      mfbca * (vvx - c1o2) + (mfaca + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcca        - c2o1 * mfbca *  vvx         +  mfaca                   * (c1o1 - vx2)              - c1o36 * oMdrho * vx2; 
	  m2 =  mfcca * c1o2 +      mfbca * (vvx + c1o2) + (mfaca + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaca = m0;
	  mfbca = m1;
	  mfcca = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcab * c1o2 +      mfbab * (vvx - c1o2) + (mfaab + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcab        - c2o1 * mfbab *  vvx         +  mfaab                  * (c1o1 - vx2)              - c1o9 * oMdrho * vx2; 
	  m2 =  mfcab * c1o2 +      mfbab * (vvx + c1o2) + (mfaab + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaab = m0;
	  mfbab = m1;
	  mfcab = m2;
	  ///////////b////////////////////////////////////////////////////////////////////////
	  m0 =  mfcbb * c1o2 +      mfbbb * (vvx - c1o2) + (mfabb + c4o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcbb        - c2o1 * mfbbb *  vvx         +  mfabb                  * (c1o1 - vx2)              - c4o9 * oMdrho * vx2; 
	  m2 =  mfcbb * c1o2 +      mfbbb * (vvx + c1o2) + (mfabb + c4o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfabb = m0;
	  mfbbb = m1;
	  mfcbb = m2;
	  ///////////b////////////////////////////////////////////////////////////////////////
	  m0 =  mfccb * c1o2 +      mfbcb * (vvx - c1o2) + (mfacb + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfccb        - c2o1 * mfbcb *  vvx         +  mfacb                  * (c1o1 - vx2)              - c1o9 * oMdrho * vx2; 
	  m2 =  mfccb * c1o2 +      mfbcb * (vvx + c1o2) + (mfacb + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfacb = m0;
	  mfbcb = m1;
	  mfccb = m2;
	  ////////////////////////////////////////////////////////////////////////////////////
	  ////////////////////////////////////////////////////////////////////////////////////
	  m0 =  mfcac * c1o2 +      mfbac * (vvx - c1o2) + (mfaac + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcac        - c2o1 * mfbac *  vvx         +  mfaac                   * (c1o1 - vx2)              - c1o36 * oMdrho * vx2; 
	  m2 =  mfcac * c1o2 +      mfbac * (vvx + c1o2) + (mfaac + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfaac = m0;
	  mfbac = m1;
	  mfcac = m2;
	  ///////////c////////////////////////////////////////////////////////////////////////
	  m0 =  mfcbc * c1o2 +      mfbbc * (vvx - c1o2) + (mfabc + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfcbc        - c2o1 * mfbbc *  vvx         +  mfabc                  * (c1o1 - vx2)              - c1o9 * oMdrho * vx2; 
	  m2 =  mfcbc * c1o2 +      mfbbc * (vvx + c1o2) + (mfabc + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfabc = m0;
	  mfbbc = m1;
	  mfcbc = m2;
	  ///////////c////////////////////////////////////////////////////////////////////////
	  m0 =  mfccc * c1o2 +      mfbcc * (vvx - c1o2) + (mfacc + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
	  m1 = -mfccc        - c2o1 * mfbcc *  vvx         +  mfacc                   * (c1o1 - vx2)              - c1o36 * oMdrho * vx2; 
	  m2 =  mfccc * c1o2 +      mfbcc * (vvx + c1o2) + (mfacc + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
	  mfacc = m0;
	  mfbcc = m1;
	  mfccc = m2;
	  ////////////////////////////////////////////////////////////////////////////////////

	  ////////////////////////////////////////////////////////////////////////////////////
	  //index 0
	  k000 = posC[k];
	  kM00 = neighborCX[k000];
	  k0M0 = neighborCY[k000];
	  k00M = neighborCZ[k000];
	  kMM0 = neighborCY[kM00];
	  kM0M = neighborCZ[kM00];
	  k0MM = neighborCZ[k0M0];
	  kMMM = neighborCZ[kMM0];
	  ////////////////////////////////////////////////////////////////////////////////////

	  ////////////////////////////////////////////////////////////////////////////////////
	  (G.g[DIR_P00])[k000] = mgcbb;
	  (G.g[DIR_M00])[kM00] = mgabb;
	  (G.g[DIR_0P0])[k000] = mgbcb;
	  (G.g[DIR_0M0])[k0M0] = mgbab;
	  (G.g[DIR_00P])[k000] = mgbbc;
	  (G.g[DIR_00M])[k00M] = mgbba;
	  ////////////////////////////////////////////////////////////////////////////////////
	  fP00dest[k000] = mfcbb;                                                                 
	  fM00dest[kM00] = mfabb;                                                               
	  f0P0dest[k000] = mfbcb;
	  f0M0dest[k0M0] = mfbab;
	  f00Pdest[k000] = mfbbc;
	  f00Mdest[k00M] = mfbba;
	  fPP0dest[k000] = mfccb;
	  fMM0dest[kMM0] = mfaab;
	  fPM0dest[k0M0] = mfcab;
	  fMP0dest[kM00] = mfacb;
	  fP0Pdest[k000] = mfcbc;
	  fM0Mdest[kM0M] = mfaba;
	  fP0Mdest[k00M] = mfcba;
	  fM0Pdest[kM00] = mfabc;
	  f0PPdest[k000] = mfbcc;
	  f0MMdest[k0MM] = mfbaa;
	  f0PMdest[k00M] = mfbca;
	  f0MPdest[k0M0] = mfbac;
	  f000dest[k000] = mfbbb;
	  fMMMdest[kMMM] = mfaaa;
	  fMMPdest[kMM0] = mfaac;
	  fMPPdest[kM00] = mfacc;
	  fMPMdest[kM0M] = mfaca;
	  fPPMdest[k00M] = mfcca;
	  fPPPdest[k000] = mfccc;
	  fPMPdest[k0M0] = mfcac;
	  fPMMdest[k0MM] = mfcaa;
	  ////////////////////////////////////////////////////////////////////////////////////
   }
}
//////////////////////////////////////////////////////////////////////////






















































