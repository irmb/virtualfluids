#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __         
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |        
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |        
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |        
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____    
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|   
//      \    \  |    |   ________________________________________________________________    
//       \    \ |    |  |  ______________________________________________________________|   
//        \    \|    |  |  |         __          __     __     __     ______      _______    
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)   
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______    
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/   
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can 
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of 
//  the License, or (at your option) any later version.
//  
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT 
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or 
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License 
//  for more details.
//  
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file scaleFC_compressible.cu
//! \ingroup GPU/GridScaling
//! \author Martin Schoenherr, Anna Wellmann
//=======================================================================================

#include "LBM/GPUHelperFunctions/ChimeraTransformation.h"
#include "LBM/GPUHelperFunctions/KernelUtilities.h"
#include "LBM/GPUHelperFunctions/ScalingUtilities.h"

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;
using namespace vf::gpu;

//////////////////////////////////////////////////////////////////////////
//! \brief Interpolate from fine to coarse
//! \details This scaling function is designed for the Cumulant K17 Kernel chimera collision kernel
//! The function is executed in the following steps:
//!

// based on scaleFC_RhoSq_comp_27
template<bool hasTurbulentViscosity> __global__ void scaleFC_compressible(
    real *distributionsCoarse,
    real *distributionsFine,
    unsigned int *neighborXcoarse,
    unsigned int *neighborYcoarse,
    unsigned int *neighborZcoarse,
    unsigned int *neighborXfine,
    unsigned int *neighborYfine,
    unsigned int *neighborZfine,
    unsigned long long numberOfLBnodesCoarse,
    unsigned long long numberOfLBnodesFine,
    bool isEvenTimestep,
    unsigned int *indicesCoarse000,
    unsigned int *indicesFineMMM,
    unsigned int numberOfInterfaceNodes,
    real omegaCoarse,
    real omegaFine,
    real* turbulentViscosityCoarse,
    real* turbulentViscosityFine,
    ICellNeighFC offsetFC)
{
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get the node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned nodeIndex = getNodeIndex();

    //////////////////////////////////////////////////////////////////////////
    //! - Return for non-interface node
    if (nodeIndex >= numberOfInterfaceNodes)
        return;

    //////////////////////////////////////////////////////////////////////////
    //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on
    //! timestep is based on the esoteric twist algorithm \ref <a
    //! href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017),
    //! DOI:10.3390/computation5020019 ]</b></a>
    //!
    Distributions27 distFine, distCoarse;
    getPointersToDistributions(distFine, distributionsFine, numberOfLBnodesFine, true);
    getPointersToDistributions(distCoarse, distributionsCoarse, numberOfLBnodesCoarse, isEvenTimestep);

    ////////////////////////////////////////////////////////////////////////////////
    //! - declare local variables for source nodes
    //!
    real eps_new = c2o1; // ratio of grid resolutions
    real omegaF  = omegaFine;
    real omegaC  = omegaCoarse;

    // zeroth and first order moments at the source nodes
    real drho_PPP, vx1_PPP, vx2_PPP, vx3_PPP;
    real drho_MPP, vx1_MPP, vx2_MPP, vx3_MPP;
    real drho_PMP, vx1_PMP, vx2_PMP, vx3_PMP;
    real drho_MMP, vx1_MMP, vx2_MMP, vx3_MMP;
    real drho_PPM, vx1_PPM, vx2_PPM, vx3_PPM;
    real drho_MPM, vx1_MPM, vx2_MPM, vx3_MPM;
    real drho_PMM, vx1_PMM, vx2_PMM, vx3_PMM;
    real drho_MMM, vx1_MMM, vx2_MMM, vx3_MMM;

    // second order moments at the source nodes
    real kxyFromfcNEQ_PPP, kyzFromfcNEQ_PPP, kxzFromfcNEQ_PPP, kxxMyyFromfcNEQ_PPP, kxxMzzFromfcNEQ_PPP;
    real kxyFromfcNEQ_MPP, kyzFromfcNEQ_MPP, kxzFromfcNEQ_MPP, kxxMyyFromfcNEQ_MPP, kxxMzzFromfcNEQ_MPP;
    real kxyFromfcNEQ_PMP, kyzFromfcNEQ_PMP, kxzFromfcNEQ_PMP, kxxMyyFromfcNEQ_PMP, kxxMzzFromfcNEQ_PMP;
    real kxyFromfcNEQ_MMP, kyzFromfcNEQ_MMP, kxzFromfcNEQ_MMP, kxxMyyFromfcNEQ_MMP, kxxMzzFromfcNEQ_MMP;
    real kxyFromfcNEQ_PPM, kyzFromfcNEQ_PPM, kxzFromfcNEQ_PPM, kxxMyyFromfcNEQ_PPM, kxxMzzFromfcNEQ_PPM;
    real kxyFromfcNEQ_MPM, kyzFromfcNEQ_MPM, kxzFromfcNEQ_MPM, kxxMyyFromfcNEQ_MPM, kxxMzzFromfcNEQ_MPM;
    real kxyFromfcNEQ_PMM, kyzFromfcNEQ_PMM, kxzFromfcNEQ_PMM, kxxMyyFromfcNEQ_PMM, kxxMzzFromfcNEQ_PMM;
    real kxyFromfcNEQ_MMM, kyzFromfcNEQ_MMM, kxzFromfcNEQ_MMM, kxxMyyFromfcNEQ_MMM, kxxMzzFromfcNEQ_MMM;

    //////////////////////////////////////////////////////////////////////////
    //! - Calculate moments for each source node 
    //!
    //////////////////////////////////////////////////////////////////////////
    // source node BSW = MMM
    //////////////////////////////////////////////////////////////////////////
    // index of the base node and its neighbors
    unsigned int k_base_000 = indicesFineMMM[nodeIndex];
    unsigned int k_base_M00 = neighborXfine [k_base_000];
    unsigned int k_base_0M0 = neighborYfine [k_base_000];
    unsigned int k_base_00M = neighborZfine [k_base_000];
    unsigned int k_base_MM0 = neighborYfine [k_base_M00];
    unsigned int k_base_M0M = neighborZfine [k_base_M00];
    unsigned int k_base_0MM = neighborZfine [k_base_0M0];
    unsigned int k_base_MMM = neighborZfine [k_base_MM0];
    //////////////////////////////////////////////////////////////////////////
    // Set neighbor indices
    unsigned int k_000 = k_base_000;
    unsigned int k_M00 = k_base_M00;
    unsigned int k_0M0 = k_base_0M0;
    unsigned int k_00M = k_base_00M;
    unsigned int k_MM0 = k_base_MM0;
    unsigned int k_M0M = k_base_M0M;
    unsigned int k_0MM = k_base_0MM;
    unsigned int k_MMM = k_base_MMM;

    if(hasTurbulentViscosity) omegaF = omegaFine/ (c1o1 + c3o1*omegaFine*turbulentViscosityFine[k_000]);

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_MMM, vx1_MMM, vx2_MMM, vx3_MMM,
        kxyFromfcNEQ_MMM, kyzFromfcNEQ_MMM, kxzFromfcNEQ_MMM, kxxMyyFromfcNEQ_MMM, kxxMzzFromfcNEQ_MMM);

    //////////////////////////////////////////////////////////////////////////
    // source node TSW = MMP
    //////////////////////////////////////////////////////////////////////////
    // Set neighbor indices - has to be recalculated for the new source node
    k_000 = k_00M;
    k_M00 = k_M0M;
    k_0M0 = k_0MM;
    k_00M = neighborZfine[k_00M];
    k_MM0 = k_MMM;
    k_M0M = neighborZfine[k_M0M];
    k_0MM = neighborZfine[k_0MM];
    k_MMM = neighborZfine[k_MMM];

    if(hasTurbulentViscosity) omegaF = omegaFine/ (c1o1 + c3o1*omegaFine*turbulentViscosityFine[k_000]);

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_MMP, vx1_MMP, vx2_MMP, vx3_MMP,
        kxyFromfcNEQ_MMP, kyzFromfcNEQ_MMP, kxzFromfcNEQ_MMP, kxxMyyFromfcNEQ_MMP, kxxMzzFromfcNEQ_MMP);

    //////////////////////////////////////////////////////////////////////////
    // source node TSE = PMP
    //////////////////////////////////////////////////////////////////////////
    // index
    k_000 = k_M00;
    k_M00 = neighborXfine[k_M00];
    k_0M0 = k_MM0;
    k_00M = k_M0M;
    k_MM0 = neighborXfine[k_MM0];
    k_M0M = neighborXfine[k_M0M];
    k_0MM = k_MMM;
    k_MMM = neighborXfine[k_MMM];

    if(hasTurbulentViscosity) omegaF = omegaFine/ (c1o1 + c3o1*omegaFine*turbulentViscosityFine[k_000]);

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_PMP, vx1_PMP, vx2_PMP, vx3_PMP,
        kxyFromfcNEQ_PMP, kyzFromfcNEQ_PMP, kxzFromfcNEQ_PMP, kxxMyyFromfcNEQ_PMP, kxxMzzFromfcNEQ_PMP);

    //////////////////////////////////////////////////////////////////////////
    // source node BSE = PMM 
    //////////////////////////////////////////////////////////////////////////
    // index
    k_00M = k_000;
    k_M0M = k_M00;
    k_0MM = k_0M0;
    k_MMM = k_MM0;
    k_000 = k_base_M00;
    k_M00 = neighborXfine[k_base_M00];
    k_0M0 = k_base_MM0;
    k_MM0 = neighborXfine[k_base_MM0];

    if(hasTurbulentViscosity) omegaF = omegaFine/ (c1o1 + c3o1*omegaFine*turbulentViscosityFine[k_000]);

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_PMM, vx1_PMM, vx2_PMM, vx3_PMM,
        kxyFromfcNEQ_PMM, kyzFromfcNEQ_PMM, kxzFromfcNEQ_PMM, kxxMyyFromfcNEQ_PMM, kxxMzzFromfcNEQ_PMM);

    //////////////////////////////////////////////////////////////////////////
    // source node BNW = MPM
    //////////////////////////////////////////////////////////////////////////
    // index of the base node and its neighbors --> indices of all source nodes
    k_base_000 = k_base_0M0;
    k_base_M00 = k_base_MM0;
    k_base_0M0 = neighborYfine[k_base_0M0];
    k_base_00M = k_base_0MM;
    k_base_MM0 = neighborYfine[k_base_MM0];
    k_base_M0M = k_base_MMM;
    k_base_0MM = neighborYfine[k_base_0MM];
    k_base_MMM = neighborYfine[k_base_MMM];
    //////////////////////////////////////////////////////////////////////////
    // index
    k_000 = k_base_000;
    k_M00 = k_base_M00;
    k_0M0 = k_base_0M0;
    k_00M = k_base_00M;
    k_MM0 = k_base_MM0;
    k_M0M = k_base_M0M;
    k_0MM = k_base_0MM;
    k_MMM = k_base_MMM;

    if(hasTurbulentViscosity) omegaF = omegaFine/ (c1o1 + c3o1*omegaFine*turbulentViscosityFine[k_000]);

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_MPM, vx1_MPM, vx2_MPM, vx3_MPM,
        kxyFromfcNEQ_MPM, kyzFromfcNEQ_MPM, kxzFromfcNEQ_MPM, kxxMyyFromfcNEQ_MPM, kxxMzzFromfcNEQ_MPM);

    //////////////////////////////////////////////////////////////////////////
    // source node TNW = MPP
    //////////////////////////////////////////////////////////////////////////
    // index
    k_000 = k_00M;
    k_M00 = k_M0M;
    k_0M0 = k_0MM;
    k_00M = neighborZfine[k_00M];
    k_MM0 = k_MMM;
    k_M0M = neighborZfine[k_M0M];
    k_0MM = neighborZfine[k_0MM];
    k_MMM = neighborZfine[k_MMM];

    if(hasTurbulentViscosity) omegaF = omegaFine/ (c1o1 + c3o1*omegaFine*turbulentViscosityFine[k_000]);
    
    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_MPP, vx1_MPP, vx2_MPP, vx3_MPP,
        kxyFromfcNEQ_MPP, kyzFromfcNEQ_MPP, kxzFromfcNEQ_MPP, kxxMyyFromfcNEQ_MPP, kxxMzzFromfcNEQ_MPP);

    //////////////////////////////////////////////////////////////////////////
    // source node TNE = PPP
    //////////////////////////////////////////////////////////////////////////
    // index
    k_000 = k_M00;
    k_M00 = neighborXfine[k_M00];
    k_0M0 = k_MM0;
    k_00M = k_M0M;
    k_MM0 = neighborXfine[k_MM0];
    k_M0M = neighborXfine[k_M0M];
    k_0MM = k_MMM;
    k_MMM = neighborXfine[k_MMM];

    if(hasTurbulentViscosity) omegaF = omegaFine/ (c1o1 + c3o1*omegaFine*turbulentViscosityFine[k_000]);

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_PPP, vx1_PPP, vx2_PPP, vx3_PPP,
        kxyFromfcNEQ_PPP, kyzFromfcNEQ_PPP, kxzFromfcNEQ_PPP, kxxMyyFromfcNEQ_PPP, kxxMzzFromfcNEQ_PPP);

    //////////////////////////////////////////////////////////////////////////
    // source node BNE = PPM
    //////////////////////////////////////////////////////////////////////////
    // index
    k_00M = k_000;
    k_M0M = k_M00;
    k_0MM = k_0M0;
    k_MMM = k_MM0;
    k_000 = k_base_M00;
    k_M00 = neighborXfine[k_base_M00];
    k_0M0 = k_base_MM0;
    k_MM0 = neighborXfine[k_base_MM0];
    
    if(hasTurbulentViscosity) omegaF = omegaFine/ (c1o1 + c3o1*omegaFine*turbulentViscosityFine[k_000]);

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_PPM, vx1_PPM, vx2_PPM, vx3_PPM,
        kxyFromfcNEQ_PPM, kyzFromfcNEQ_PPM, kxzFromfcNEQ_PPM, kxxMyyFromfcNEQ_PPM, kxxMzzFromfcNEQ_PPM);

    //////////////////////////////////////////////////////////////////////////
    //! - Calculate coefficients for polynomial interpolation
    //!
    // example: a_110: derivation in x and y direction
    real a_000, a_100, a_010, a_001, a_200, a_020, a_002, a_110, a_101, a_011;
    real b_000, b_100, b_010, b_001, b_200, b_020, b_002, b_110, b_101, b_011;
    real c_000, c_100, c_010, c_001, c_200, c_020, c_002, c_110, c_101, c_011;
    real d_000, d_100, d_010, d_001, d_110, d_101, d_011;

    a_000 = c1o64 * (
            c2o1 * (
            ((kxyFromfcNEQ_MMM - kxyFromfcNEQ_PPP) + (kxyFromfcNEQ_MMP - kxyFromfcNEQ_PPM)) + ((kxyFromfcNEQ_PMM - kxyFromfcNEQ_MPP) + (kxyFromfcNEQ_PMP - kxyFromfcNEQ_MPM)) + 
            ((kxzFromfcNEQ_MMM - kxzFromfcNEQ_PPP) + (kxzFromfcNEQ_PPM - kxzFromfcNEQ_MMP)) + ((kxzFromfcNEQ_PMM - kxzFromfcNEQ_MPP) + (kxzFromfcNEQ_MPM - kxzFromfcNEQ_PMP)) + 
            ((vx2_PPP + vx2_MMM) + (vx2_PPM + vx2_MMP)) - ((vx2_MPP + vx2_PMM) + (vx2_MPM + vx2_PMP)) + 
            ((vx3_PPP + vx3_MMM) - (vx3_PPM + vx3_MMP)) + ((vx3_PMP + vx3_MPM) - (vx3_MPP + vx3_PMM))) + 
            c8o1 * (((vx1_PPP + vx1_MMM) + (vx1_PPM + vx1_MMP)) + ((vx1_MPP + vx1_PMM) + (vx1_PMP + vx1_MPM))) +
            ((kxxMyyFromfcNEQ_MMM - kxxMyyFromfcNEQ_PPP) + (kxxMyyFromfcNEQ_MMP - kxxMyyFromfcNEQ_PPM)) + 
            ((kxxMyyFromfcNEQ_MPP - kxxMyyFromfcNEQ_PMM) + (kxxMyyFromfcNEQ_MPM - kxxMyyFromfcNEQ_PMP)) +
            ((kxxMzzFromfcNEQ_MMM - kxxMzzFromfcNEQ_PPP) + (kxxMzzFromfcNEQ_MMP - kxxMzzFromfcNEQ_PPM)) + 
            ((kxxMzzFromfcNEQ_MPP - kxxMzzFromfcNEQ_PMM) + (kxxMzzFromfcNEQ_MPM - kxxMzzFromfcNEQ_PMP)));
    b_000 = c1o64 * (
            c2o1 * (
            ((kxxMyyFromfcNEQ_PPP - kxxMyyFromfcNEQ_MMM) + (kxxMyyFromfcNEQ_PPM - kxxMyyFromfcNEQ_MMP)) + 
            ((kxxMyyFromfcNEQ_MPP - kxxMyyFromfcNEQ_PMM) + (kxxMyyFromfcNEQ_MPM - kxxMyyFromfcNEQ_PMP)) + 
            ((kxyFromfcNEQ_MMM - kxyFromfcNEQ_PPP) + (kxyFromfcNEQ_MMP - kxyFromfcNEQ_PPM)) + 
            ((kxyFromfcNEQ_MPP - kxyFromfcNEQ_PMM) + (kxyFromfcNEQ_MPM - kxyFromfcNEQ_PMP)) + 
            ((kyzFromfcNEQ_MMM - kyzFromfcNEQ_PPP) + (kyzFromfcNEQ_PPM - kyzFromfcNEQ_MMP)) + 
            ((kyzFromfcNEQ_PMM - kyzFromfcNEQ_MPP) + (kyzFromfcNEQ_MPM - kyzFromfcNEQ_PMP)) + 
            ((vx1_PPP + vx1_MMM) + (vx1_PPM + vx1_MMP)) - ((vx1_MPM + vx1_MPP) + (vx1_PMM + vx1_PMP)) + 
            ((vx3_PPP + vx3_MMM) - (vx3_PPM + vx3_MMP)) + ((vx3_MPP + vx3_PMM) - (vx3_MPM + vx3_PMP))) + 
            c8o1 * (((vx2_PPP + vx2_MMM) + (vx2_PPM + vx2_MMP)) + ((vx2_MPP + vx2_PMM) + (vx2_MPM + vx2_PMP))) + 
            ((kxxMzzFromfcNEQ_MMM - kxxMzzFromfcNEQ_PPP) + (kxxMzzFromfcNEQ_MMP - kxxMzzFromfcNEQ_PPM)) +
            ((kxxMzzFromfcNEQ_PMM - kxxMzzFromfcNEQ_MPP) + (kxxMzzFromfcNEQ_PMP - kxxMzzFromfcNEQ_MPM)));
    c_000 = c1o64 * ( 
            c2o1 * (
            ((kxxMzzFromfcNEQ_PPP - kxxMzzFromfcNEQ_MMM) + (kxxMzzFromfcNEQ_MMP - kxxMzzFromfcNEQ_PPM)) + 
            ((kxxMzzFromfcNEQ_MPP - kxxMzzFromfcNEQ_PMM) + (kxxMzzFromfcNEQ_PMP - kxxMzzFromfcNEQ_MPM)) + 
            ((kxzFromfcNEQ_MMM - kxzFromfcNEQ_PPP) + (kxzFromfcNEQ_MMP - kxzFromfcNEQ_PPM)) + 
            ((kxzFromfcNEQ_MPP - kxzFromfcNEQ_PMM) + (kxzFromfcNEQ_MPM - kxzFromfcNEQ_PMP)) + 
            ((kyzFromfcNEQ_MMM - kyzFromfcNEQ_PPP) + (kyzFromfcNEQ_MMP - kyzFromfcNEQ_PPM)) + 
            ((kyzFromfcNEQ_PMM - kyzFromfcNEQ_MPP) + (kyzFromfcNEQ_PMP - kyzFromfcNEQ_MPM)) + 
            ((vx1_PPP + vx1_MMM) - (vx1_MMP + vx1_PPM)) + ((vx1_MPM + vx1_PMP) - (vx1_MPP + vx1_PMM)) + 
            ((vx2_PPP + vx2_MMM) - (vx2_MMP + vx2_PPM)) + ((vx2_MPP + vx2_PMM) - (vx2_MPM + vx2_PMP))) + 
            c8o1 * (((vx3_PPP + vx3_MMM) + (vx3_PPM + vx3_MMP)) + ((vx3_PMM + vx3_MPP) + (vx3_PMP + vx3_MPM))) +
            ((kxxMyyFromfcNEQ_MMM - kxxMyyFromfcNEQ_PPP) + (kxxMyyFromfcNEQ_PPM - kxxMyyFromfcNEQ_MMP)) + 
            ((kxxMyyFromfcNEQ_PMM - kxxMyyFromfcNEQ_MPP) + (kxxMyyFromfcNEQ_MPM - kxxMyyFromfcNEQ_PMP)));

    a_100 = c1o4 * (((vx1_PPP - vx1_MMM) + (vx1_PPM - vx1_MMP)) + ((vx1_PMM - vx1_MPP) + (vx1_PMP - vx1_MPM)));
    b_100 = c1o4 * (((vx2_PPP - vx2_MMM) + (vx2_PPM - vx2_MMP)) + ((vx2_PMM - vx2_MPP) + (vx2_PMP - vx2_MPM)));
    c_100 = c1o4 * (((vx3_PPP - vx3_MMM) + (vx3_PPM - vx3_MMP)) + ((vx3_PMM - vx3_MPP) + (vx3_PMP - vx3_MPM)));

    a_200 = c1o16 * ( 
            c2o1 * (
            ((vx2_PPP + vx2_MMM) + (vx2_PPM - vx2_MPP)) + ((vx2_MMP - vx2_PMM) - (vx2_MPM + vx2_PMP)) + 
            ((vx3_PPP + vx3_MMM) - (vx3_PPM + vx3_MPP)) + ((vx3_MPM + vx3_PMP) - (vx3_MMP + vx3_PMM))) + 
            ((kxxMyyFromfcNEQ_PPP - kxxMyyFromfcNEQ_MMM) + (kxxMyyFromfcNEQ_PPM - kxxMyyFromfcNEQ_MMP)) + 
            ((kxxMyyFromfcNEQ_PMM - kxxMyyFromfcNEQ_MPP) + (kxxMyyFromfcNEQ_PMP - kxxMyyFromfcNEQ_MPM)) + 
            ((kxxMzzFromfcNEQ_PPP - kxxMzzFromfcNEQ_MMM) + (kxxMzzFromfcNEQ_PPM - kxxMzzFromfcNEQ_MMP)) + 
            ((kxxMzzFromfcNEQ_PMM - kxxMzzFromfcNEQ_MPP) + (kxxMzzFromfcNEQ_PMP - kxxMzzFromfcNEQ_MPM)));
    b_200 = c1o8 * (
            c2o1 * (
            -((vx1_PPP + vx1_MMM) + (vx1_PPM + vx1_MMP)) + ((vx1_MPP + vx1_PMM) + (vx1_MPM + vx1_PMP))) +
            ((kxyFromfcNEQ_PPP - kxyFromfcNEQ_MMM) + (kxyFromfcNEQ_PPM - kxyFromfcNEQ_MMP)) + 
            ((kxyFromfcNEQ_PMM - kxyFromfcNEQ_MPP) + (kxyFromfcNEQ_PMP - kxyFromfcNEQ_MPM)));
    c_200 = c1o8 * (
            c2o1 * (
            ((vx1_PPM + vx1_MMP) - (vx1_PPP + vx1_MMM)) + ((vx1_MPP + vx1_PMM) - (vx1_MPM + vx1_PMP))) +
            ((kxzFromfcNEQ_PPP - kxzFromfcNEQ_MMM) + (kxzFromfcNEQ_PPM - kxzFromfcNEQ_MMP)) + 
            ((kxzFromfcNEQ_PMM - kxzFromfcNEQ_MPP) + (kxzFromfcNEQ_PMP - kxzFromfcNEQ_MPM)));

    a_010 = c1o4 * (((vx1_PPP - vx1_MMM) + (vx1_PPM - vx1_MMP)) + ((vx1_MPP - vx1_PMM) + (vx1_MPM - vx1_PMP)));
    b_010 = c1o4 * (((vx2_PPP - vx2_MMM) + (vx2_PPM - vx2_MMP)) + ((vx2_MPP - vx2_PMM) + (vx2_MPM - vx2_PMP)));
    c_010 = c1o4 * (((vx3_PPP - vx3_MMM) + (vx3_PPM - vx3_MMP)) + ((vx3_MPP - vx3_PMM) + (vx3_MPM - vx3_PMP)));

    a_020 = c1o8 * (
            c2o1 * (-((vx2_PPP + vx2_MMM) + (vx2_MMP + vx2_PPM)) + ((vx2_MPP + vx2_PMM) + (vx2_MPM + vx2_PMP))) +
            ((kxyFromfcNEQ_PPP - kxyFromfcNEQ_MMM) + (kxyFromfcNEQ_PPM - kxyFromfcNEQ_MMP)) + 
            ((kxyFromfcNEQ_MPP - kxyFromfcNEQ_PMM) + (kxyFromfcNEQ_MPM - kxyFromfcNEQ_PMP)));
    b_020 = c1o16 * (
            c2o1 * (
            ((kxxMyyFromfcNEQ_MMM - kxxMyyFromfcNEQ_PPP) + (kxxMyyFromfcNEQ_MMP - kxxMyyFromfcNEQ_PPM)) +
            ((kxxMyyFromfcNEQ_PMM - kxxMyyFromfcNEQ_MPP) + (kxxMyyFromfcNEQ_PMP - kxxMyyFromfcNEQ_MPM)) +
            ((vx1_PPP + vx1_MMM) + (vx1_PPM + vx1_MMP)) - ((vx1_MPP + vx1_PMM) + (vx1_PMP + vx1_MPM)) + 
            ((vx3_PPP + vx3_MMM) - (vx3_PPM + vx3_MMP)) + ((vx3_MPP + vx3_PMM) - (vx3_MPM + vx3_PMP))) +
            ((kxxMzzFromfcNEQ_PPP - kxxMzzFromfcNEQ_MMM) + (kxxMzzFromfcNEQ_PPM - kxxMzzFromfcNEQ_MMP)) + 
            ((kxxMzzFromfcNEQ_MPP - kxxMzzFromfcNEQ_PMM) + (kxxMzzFromfcNEQ_MPM - kxxMzzFromfcNEQ_PMP)));
    c_020 = c1o8 * (
            c2o1 * (((vx2_MMP + vx2_PPM) - (vx2_PPP + vx2_MMM)) + ((vx2_PMP + vx2_MPM) - (vx2_MPP + vx2_PMM))) +
            ((kyzFromfcNEQ_PPP - kyzFromfcNEQ_MMM) + (kyzFromfcNEQ_PPM - kyzFromfcNEQ_MMP)) +
            ((kyzFromfcNEQ_MPP - kyzFromfcNEQ_PMM) + (kyzFromfcNEQ_MPM - kyzFromfcNEQ_PMP)));

    a_001 = c1o4 * (((vx1_PPP - vx1_MMM) + (vx1_MMP - vx1_PPM)) + ((vx1_MPP - vx1_PMM) + (vx1_PMP - vx1_MPM)));
    b_001 = c1o4 * (((vx2_PPP - vx2_MMM) + (vx2_MMP - vx2_PPM)) + ((vx2_MPP - vx2_PMM) + (vx2_PMP - vx2_MPM)));
    c_001 = c1o4 * (((vx3_PPP - vx3_MMM) + (vx3_MMP - vx3_PPM)) + ((vx3_MPP - vx3_PMM) + (vx3_PMP - vx3_MPM)));

    a_002 = c1o8 * (
            c2o1 * (((vx3_PPM + vx3_MMP) - (vx3_PPP + vx3_MMM)) + ((vx3_MPP + vx3_PMM) - (vx3_PMP + vx3_MPM))) +
                    ((kxzFromfcNEQ_PPP - kxzFromfcNEQ_MMM) + (kxzFromfcNEQ_MMP - kxzFromfcNEQ_PPM)) +
                    ((kxzFromfcNEQ_PMP - kxzFromfcNEQ_MPM) + (kxzFromfcNEQ_MPP - kxzFromfcNEQ_PMM)));
    b_002 = c1o8 * (
            c2o1 * (((vx3_PPM + vx3_MMP) - (vx3_PPP + vx3_MMM)) + ((vx3_MPM + vx3_PMP) - (vx3_PMM + vx3_MPP))) + 
                    ((kyzFromfcNEQ_PPP - kyzFromfcNEQ_MMM) + (kyzFromfcNEQ_MMP - kyzFromfcNEQ_PPM)) + 
                    ((kyzFromfcNEQ_PMP - kyzFromfcNEQ_MPM) + (kyzFromfcNEQ_MPP - kyzFromfcNEQ_PMM)));
    c_002 = c1o16 * (
            c2o1 * (
            ((kxxMzzFromfcNEQ_MMM - kxxMzzFromfcNEQ_PPP) + (kxxMzzFromfcNEQ_PPM - kxxMzzFromfcNEQ_MMP)) + 
            ((kxxMzzFromfcNEQ_MPM - kxxMzzFromfcNEQ_PMP) + (kxxMzzFromfcNEQ_PMM - kxxMzzFromfcNEQ_MPP)) + 
            ((vx1_PPP + vx1_MMM) - (vx1_MMP + vx1_PPM)) + ((vx1_MPM + vx1_PMP) - (vx1_PMM + vx1_MPP)) + 
            ((vx2_PPP + vx2_MMM) - (vx2_MMP + vx2_PPM)) + ((vx2_PMM + vx2_MPP) - (vx2_MPM + vx2_PMP))) + 
            ((kxxMyyFromfcNEQ_PPP - kxxMyyFromfcNEQ_MMM) + (kxxMyyFromfcNEQ_MMP - kxxMyyFromfcNEQ_PPM)) +
            ((kxxMyyFromfcNEQ_PMP - kxxMyyFromfcNEQ_MPM) + (kxxMyyFromfcNEQ_MPP - kxxMyyFromfcNEQ_PMM)));

    a_110 = c1o2 * (((vx1_PPP + vx1_MMM) + (vx1_MMP + vx1_PPM)) - ((vx1_MPM + vx1_PMP) + (vx1_PMM + vx1_MPP)));
    b_110 = c1o2 * (((vx2_PPP + vx2_MMM) + (vx2_MMP + vx2_PPM)) - ((vx2_MPM + vx2_PMP) + (vx2_PMM + vx2_MPP)));
    c_110 = c1o2 * (((vx3_PPP + vx3_MMM) + (vx3_MMP + vx3_PPM)) - ((vx3_MPM + vx3_PMP) + (vx3_PMM + vx3_MPP)));

    a_101 = c1o2 * (((vx1_PPP + vx1_MMM) - (vx1_MMP + vx1_PPM)) + ((vx1_MPM + vx1_PMP) - (vx1_PMM + vx1_MPP)));
    b_101 = c1o2 * (((vx2_PPP + vx2_MMM) - (vx2_MMP + vx2_PPM)) + ((vx2_MPM + vx2_PMP) - (vx2_PMM + vx2_MPP)));
    c_101 = c1o2 * (((vx3_PPP + vx3_MMM) - (vx3_MMP + vx3_PPM)) + ((vx3_MPM + vx3_PMP) - (vx3_PMM + vx3_MPP)));
    
    a_011 = c1o2 * (((vx1_PPP + vx1_MMM) - (vx1_MMP + vx1_PPM)) + ((vx1_PMM + vx1_MPP) - (vx1_MPM + vx1_PMP)));
    b_011 = c1o2 * (((vx2_PPP + vx2_MMM) - (vx2_MMP + vx2_PPM)) + ((vx2_PMM + vx2_MPP) - (vx2_MPM + vx2_PMP)));
    c_011 = c1o2 * (((vx3_PPP + vx3_MMM) - (vx3_MMP + vx3_PPM)) + ((vx3_PMM + vx3_MPP) - (vx3_MPM + vx3_PMP)));

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    
    real kxyAverage    = c0o1;
    real kyzAverage    = c0o1;
    real kxzAverage    = c0o1;
    real kxxMyyAverage = c0o1;
    real kxxMzzAverage = c0o1;

    ////////////////////////////////////////////////////////////////////////////////
    //! - Set the relative position of the offset cell {-1, 0, 1}
    //!
    real xoff    = offsetFC.x[nodeIndex];
    real yoff    = offsetFC.y[nodeIndex];
    real zoff    = offsetFC.z[nodeIndex];
     
    real xoff_sq = xoff * xoff;
    real yoff_sq = yoff * yoff;
    real zoff_sq = zoff * zoff;

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //!- Calculate coefficients for the polynomial interpolation of the pressure
    //! 
    real LaplaceRho = 
        ((xoff != c0o1) || (yoff != c0o1) || (zoff != c0o1))
        ? c0o1 : -c3o1 * (a_100 * a_100 + b_010 * b_010 + c_001 * c_001) - c6o1 * (b_100 * a_010 + c_100 * a_001 + c_010 * b_001);
    d_000 =  c1o8 * ((((drho_PPP + drho_MMM) + (drho_PPM + drho_MMP)) + ((drho_PMM + drho_MPP) + (drho_PMP + drho_MPM))) - c2o1 * LaplaceRho);
    d_100 = c1o4 * (((drho_PPP - drho_MMM) + (drho_PPM - drho_MMP)) + ((drho_PMM - drho_MPP) + (drho_PMP - drho_MPM)));
    d_010 = c1o4 * (((drho_PPP - drho_MMM) + (drho_PPM - drho_MMP)) + ((drho_MPP - drho_PMM) + (drho_MPM - drho_PMP)));
    d_001 = c1o4 * (((drho_PPP - drho_MMM) + (drho_MMP - drho_PPM)) + ((drho_MPP - drho_PMM) + (drho_PMP - drho_MPM)));
    d_110 = c1o2 * (((drho_PPP + drho_MMM) + (drho_PPM + drho_MMP)) - ((drho_PMM + drho_MPP) + (drho_PMP + drho_MPM)));
    d_101 = c1o2 * (((drho_PPP + drho_MMM) - (drho_PPM + drho_MMP)) + ((drho_PMP + drho_MPM) - (drho_PMM + drho_MPP)));
    d_011 = c1o2 * (((drho_PPP + drho_MMM) - (drho_PPM + drho_MMP)) + ((drho_PMM + drho_MPP) - (drho_PMP + drho_MPM)));


    //////////////////////////////////////////////////////////////////////////
    //! - Extrapolation for refinement in to the wall (polynomial coefficients)
    //!
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //
    // x------x
    // |      |
    // |   ---+--->X
    // |      |  \
    // x------x   \
    //          offset-vector
    //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    a_000 = a_000 + xoff * a_100 + yoff * a_010 + zoff * a_001 + xoff_sq * a_200 + yoff_sq * a_020 + zoff_sq * a_002 +
            xoff * yoff * a_110 + xoff * zoff * a_101 + yoff * zoff * a_011;
    a_100 = a_100 + c2o1 * xoff * a_200 + yoff * a_110 + zoff * a_101;
    a_010 = a_010 + c2o1 * yoff * a_020 + xoff * a_110 + zoff * a_011;
    a_001 = a_001 + c2o1 * zoff * a_002 + xoff * a_101 + yoff * a_011;
    b_000 = b_000 + xoff * b_100 + yoff * b_010 + zoff * b_001 + xoff_sq * b_200 + yoff_sq * b_020 + zoff_sq * b_002 +
            xoff * yoff * b_110 + xoff * zoff * b_101 + yoff * zoff * b_011;
    b_100 = b_100 + c2o1 * xoff * b_200 + yoff * b_110 + zoff * b_101;
    b_010 = b_010 + c2o1 * yoff * b_020 + xoff * b_110 + zoff * b_011;
    b_001 = b_001 + c2o1 * zoff * b_002 + xoff * b_101 + yoff * b_011;
    c_000 = c_000 + xoff * c_100 + yoff * c_010 + zoff * c_001 + xoff_sq * c_200 + yoff_sq * c_020 + zoff_sq * c_002 +
            xoff * yoff * c_110 + xoff * zoff * c_101 + yoff * zoff * c_011;
    c_100 = c_100 + c2o1 * xoff * c_200 + yoff * c_110 + zoff * c_101;
    c_010 = c_010 + c2o1 * yoff * c_020 + xoff * c_110 + zoff * c_011;
    c_001 = c_001 + c2o1 * zoff * c_002 + xoff * c_101 + yoff * c_011;
    d_000 = d_000 + xoff * d_100 + yoff * d_010 + zoff * d_001 + 
            xoff * yoff * d_110 + xoff * zoff * d_101 + yoff * zoff * d_011;

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Set all moments to zero
    //!
    real m_111 = c0o1;
    real m_211 = c0o1;
    real m_011 = c0o1;
    real m_121 = c0o1;
    real m_101 = c0o1;
    real m_112 = c0o1;
    real m_110 = c0o1;
    real m_221 = c0o1;
    real m_001 = c0o1;
    real m_201 = c0o1;
    real m_021 = c0o1;
    real m_212 = c0o1;
    real m_010 = c0o1;
    real m_210 = c0o1;
    real m_012 = c0o1;
    real m_122 = c0o1;
    real m_100 = c0o1;
    real m_120 = c0o1;
    real m_102 = c0o1;
    real m_222 = c0o1;
    real m_022 = c0o1;
    real m_202 = c0o1;
    real m_002 = c0o1;
    real m_220 = c0o1;
    real m_020 = c0o1;
    real m_200 = c0o1;
    real m_000 = c0o1;

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Define aliases to use the same variable for the distributions (f's):
    //!
    real& f_000 = m_111;
    real& f_P00 = m_211;
    real& f_M00 = m_011;
    real& f_0P0 = m_121;
    real& f_0M0 = m_101;
    real& f_00P = m_112;
    real& f_00M = m_110;
    real& f_PP0 = m_221;
    real& f_MM0 = m_001;
    real& f_PM0 = m_201;
    real& f_MP0 = m_021;
    real& f_P0P = m_212;
    real& f_M0M = m_010;
    real& f_P0M = m_210;
    real& f_M0P = m_012;
    real& f_0PP = m_122;
    real& f_0MM = m_100;
    real& f_0PM = m_120;
    real& f_0MP = m_102;
    real& f_PPP = m_222;
    real& f_MPP = m_022;
    real& f_PMP = m_202;
    real& f_MMP = m_002;
    real& f_PPM = m_220;
    real& f_MPM = m_020;
    real& f_PMM = m_200;
    real& f_MMM = m_000;

    ////////////////////////////////////////////////////////////////////////////////
    //! - Declare local variables for destination nodes
    //!
    real vvx, vvy, vvz, vx_sq, vy_sq, vz_sq;
    real mxxPyyPzz, mxxMyy, mxxMzz, mxxyPyzz, mxxyMyzz, mxxzPyyz, mxxzMyyz, mxyyPxzz, mxyyMxzz;
    real useNEQ = c1o1; // zero; //one;   //.... one = on ..... zero = off
    real press;
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //
    // Position Coarse 0., 0., 0.
    //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // x = 0.;
    // y = 0.;
    // z = 0.;
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // index of the destination node and its neighbors
    k_000 = indicesCoarse000[nodeIndex];
    k_M00 = neighborXcoarse [k_000];
    k_0M0 = neighborYcoarse [k_000];
    k_00M = neighborZcoarse [k_000];
    k_MM0 = neighborYcoarse [k_M00];
    k_M0M = neighborZcoarse [k_M00];
    k_0MM = neighborZcoarse [k_0M0];
    k_MMM = neighborZcoarse [k_MM0];
    ////////////////////////////////////////////////////////////////////////////////////

    if(hasTurbulentViscosity) omegaC = omegaCoarse / (c1o1 + c3o1*omegaCoarse*turbulentViscosityCoarse[k_000]);

    ////////////////////////////////////////////////////////////////////////////////
    //! - Set macroscopic values on destination node (zeroth and first order moments)
    //!
    press = d_000;
    vvx   = a_000;
    vvy   = b_000;
    vvz   = c_000;

    m_000 = press; // m_000 is press, if drho is interpolated directly

    vx_sq = vvx * vvx;
    vy_sq = vvy * vvy;
    vz_sq = vvz * vvz;

    ////////////////////////////////////////////////////////////////////////////////
    //! - Set moments (second to sixth order) on destination node
    //!
    // linear combinations for second order moments
    mxxPyyPzz = m_000;

    mxxMyy = -c2o3 * ((a_100 - b_010) + kxxMyyAverage) * eps_new / omegaC * (c1o1 + press);
    mxxMzz = -c2o3 * ((a_100 - c_001) + kxxMzzAverage) * eps_new / omegaC * (c1o1 + press);

    m_011 = -c1o3 * ((b_001 + c_010) + kyzAverage) * eps_new / omegaC * (c1o1 + press);
    m_101 = -c1o3 * ((a_001 + c_100) + kxzAverage) * eps_new / omegaC * (c1o1 + press);
    m_110 = -c1o3 * ((a_010 + b_100) + kxyAverage) * eps_new / omegaC * (c1o1 + press);

    m_200 = c1o3 * (        mxxMyy +        mxxMzz + mxxPyyPzz) * useNEQ;
    m_020 = c1o3 * (-c2o1 * mxxMyy +        mxxMzz + mxxPyyPzz) * useNEQ;
    m_002 = c1o3 * (        mxxMyy - c2o1 * mxxMzz + mxxPyyPzz) * useNEQ;

    // linear combinations for third order moments
    m_111 = c0o1;

    mxxyPyzz = c0o1;
    mxxyMyzz = c0o1;
    mxxzPyyz = c0o1;
    mxxzMyyz = c0o1;
    mxyyPxzz = c0o1;
    mxyyMxzz = c0o1;

    m_210 = ( mxxyMyzz + mxxyPyzz) * c1o2;
    m_012 = (-mxxyMyzz + mxxyPyzz) * c1o2;
    m_201 = ( mxxzMyyz + mxxzPyyz) * c1o2;
    m_021 = (-mxxzMyyz + mxxzPyyz) * c1o2;
    m_120 = ( mxyyMxzz + mxyyPxzz) * c1o2;
    m_102 = (-mxyyMxzz + mxyyPxzz) * c1o2;

    // fourth order moments
    m_022 = m_000 * c1o9;
    m_202 = m_022;
    m_220 = m_022;

    // fifth order moments

    // sixth order moments
    m_222 = m_000 * c1o27;

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Chimera transform from central moments to well conditioned distributions as defined in Appendix J in
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
    //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a> see also Eq. (88)-(96) in <a
    //! href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040
    //! ]</b></a>
    //!
    ////////////////////////////////////////////////////////////////////////////////////
    // X - Dir
    backwardInverseChimeraWithK(m_000, m_100, m_200, vvx, vx_sq, c1o1, c1o1);
    backwardChimera(            m_010, m_110, m_210, vvx, vx_sq);
    backwardInverseChimeraWithK(m_020, m_120, m_220, vvx, vx_sq, c3o1, c1o3);
    backwardChimera(            m_001, m_101, m_201, vvx, vx_sq);
    backwardChimera(            m_011, m_111, m_211, vvx, vx_sq);
    backwardChimera(            m_021, m_121, m_221, vvx, vx_sq);
    backwardInverseChimeraWithK(m_002, m_102, m_202, vvx, vx_sq, c3o1, c1o3);
    backwardChimera(            m_012, m_112, m_212, vvx, vx_sq);
    backwardInverseChimeraWithK(m_022, m_122, m_222, vvx, vx_sq, c9o1, c1o9);

    ////////////////////////////////////////////////////////////////////////////////////
    // Y - Dir
    backwardInverseChimeraWithK(m_000, m_010, m_020, vvy, vy_sq, c6o1, c1o6);
    backwardChimera(            m_001, m_011, m_021, vvy, vy_sq);
    backwardInverseChimeraWithK(m_002, m_012, m_022, vvy, vy_sq, c18o1, c1o18);
    backwardInverseChimeraWithK(m_100, m_110, m_120, vvy, vy_sq, c3o2, c2o3);
    backwardChimera(            m_101, m_111, m_121, vvy, vy_sq);
    backwardInverseChimeraWithK(m_102, m_112, m_122, vvy, vy_sq, c9o2, c2o9);
    backwardInverseChimeraWithK(m_200, m_210, m_220, vvy, vy_sq, c6o1, c1o6);
    backwardChimera(            m_201, m_211, m_221, vvy, vy_sq);
    backwardInverseChimeraWithK(m_202, m_212, m_222, vvy, vy_sq, c18o1, c1o18);

    ////////////////////////////////////////////////////////////////////////////////////
    // Z - Dir
    backwardInverseChimeraWithK(m_000, m_001, m_002, vvz, vz_sq, c36o1, c1o36);
    backwardInverseChimeraWithK(m_010, m_011, m_012, vvz, vz_sq, c9o1,  c1o9);
    backwardInverseChimeraWithK(m_020, m_021, m_022, vvz, vz_sq, c36o1, c1o36);
    backwardInverseChimeraWithK(m_100, m_101, m_102, vvz, vz_sq, c9o1,  c1o9);
    backwardInverseChimeraWithK(m_110, m_111, m_112, vvz, vz_sq, c9o4,  c4o9);
    backwardInverseChimeraWithK(m_120, m_121, m_122, vvz, vz_sq, c9o1,  c1o9);
    backwardInverseChimeraWithK(m_200, m_201, m_202, vvz, vz_sq, c36o1, c1o36);
    backwardInverseChimeraWithK(m_210, m_211, m_212, vvz, vz_sq, c9o1,  c1o9);
    backwardInverseChimeraWithK(m_220, m_221, m_222, vvz, vz_sq, c36o1, c1o36);

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Write distributions: style of reading and writing the distributions from/to
    //! stored arrays dependent on timestep is based on the esoteric twist algorithm
    //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017),
    //! DOI:10.3390/computation5020019 ]</b></a>
    //!
    (distCoarse.f[DIR_000])[k_000] = f_000;
    (distCoarse.f[DIR_P00])[k_000] = f_P00;
    (distCoarse.f[DIR_M00])[k_M00] = f_M00;
    (distCoarse.f[DIR_0P0])[k_000] = f_0P0;
    (distCoarse.f[DIR_0M0])[k_0M0] = f_0M0;
    (distCoarse.f[DIR_00P])[k_000] = f_00P;
    (distCoarse.f[DIR_00M])[k_00M] = f_00M;
    (distCoarse.f[DIR_PP0])[k_000] = f_PP0;
    (distCoarse.f[DIR_MM0])[k_MM0] = f_MM0;
    (distCoarse.f[DIR_PM0])[k_0M0] = f_PM0;
    (distCoarse.f[DIR_MP0])[k_M00] = f_MP0;
    (distCoarse.f[DIR_P0P])[k_000] = f_P0P;
    (distCoarse.f[DIR_M0M])[k_M0M] = f_M0M;
    (distCoarse.f[DIR_P0M])[k_00M] = f_P0M;
    (distCoarse.f[DIR_M0P])[k_M00] = f_M0P;
    (distCoarse.f[DIR_0PP])[k_000] = f_0PP;
    (distCoarse.f[DIR_0MM])[k_0MM] = f_0MM;
    (distCoarse.f[DIR_0PM])[k_00M] = f_0PM;
    (distCoarse.f[DIR_0MP])[k_0M0] = f_0MP;
    (distCoarse.f[DIR_PPP])[k_000] = f_PPP;
    (distCoarse.f[DIR_MPP])[k_M00] = f_MPP;
    (distCoarse.f[DIR_PMP])[k_0M0] = f_PMP;
    (distCoarse.f[DIR_MMP])[k_MM0] = f_MMP;
    (distCoarse.f[DIR_PPM])[k_00M] = f_PPM;
    (distCoarse.f[DIR_MPM])[k_M0M] = f_MPM;
    (distCoarse.f[DIR_PMM])[k_0MM] = f_PMM;
    (distCoarse.f[DIR_MMM])[k_MMM] = f_MMM;
    ////////////////////////////////////////////////////////////////////////////////////
}

template __global__ void scaleFC_compressible<true>( real *distributionsCoarse, real *distributionsFine, unsigned int *neighborXcoarse, unsigned int *neighborYcoarse, unsigned int *neighborZcoarse, unsigned int *neighborXfine, unsigned int *neighborYfine, unsigned int *neighborZfine, unsigned long long numberOfLBnodesCoarse, unsigned long long numberOfLBnodesFine, bool isEvenTimestep, unsigned int *indicesCoarse000, unsigned int *indicesFineMMM, unsigned int numberOfInterfaceNodes, real omegaCoarse, real omegaFine, real* turbulentViscosityCoarse, real* turbulentViscosityFine, ICellNeighFC offsetFC);

template __global__ void scaleFC_compressible<false>( real *distributionsCoarse, real *distributionsFine, unsigned int *neighborXcoarse, unsigned int *neighborYcoarse, unsigned int *neighborZcoarse, unsigned int *neighborXfine, unsigned int *neighborYfine, unsigned int *neighborZfine, unsigned long long numberOfLBnodesCoarse, unsigned long long numberOfLBnodesFine, bool isEvenTimestep, unsigned int *indicesCoarse000, unsigned int *indicesFineMMM, unsigned int numberOfInterfaceNodes, real omegaCoarse, real omegaFine, real* turbulentViscosityCoarse, real* turbulentViscosityFine, ICellNeighFC offsetFC);