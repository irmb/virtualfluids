#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __         
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |        
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |        
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |        
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____    
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|   
//      \    \  |    |   ________________________________________________________________    
//       \    \ |    |  |  ______________________________________________________________|   
//        \    \|    |  |  |         __          __     __     __     ______      _______    
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)   
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______    
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/   
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can 
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of 
//  the License, or (at your option) any later version.
//  
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT 
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or 
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License 
//  for more details.
//  
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file scaleFC_compressible.cu
//! \ingroup GPU/GridScaling
//! \author Martin Schoenherr, Anna Wellmann
//=======================================================================================

#include "Kernel/Utilities/DistributionHelper.cuh"
#include "Kernel/Utilities/ChimeraTransformation.h"
#include "Kernel/Utilities/ScalingHelperFunctions.h"

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;

//////////////////////////////////////////////////////////////////////////
//! \brief Interpolate from fine to coarse
//! \details This scaling function is designed for the Cumulant K17 Kernel chimera collision kernel
//! The function is executed in the following steps:
//!

// based on scaleFC_RhoSq_comp_27
__global__ void scaleFC_compressible(
    real *distributionsCoarse,
    real *distributionsFine,
    unsigned int *neighborXcoarse,
    unsigned int *neighborYcoarse,
    unsigned int *neighborZcoarse,
    unsigned int *neighborXfine,
    unsigned int *neighborYfine,
    unsigned int *neighborZfine,
    unsigned int numberOfLBnodesCoarse,
    unsigned int numberOfLBnodesFine,
    bool isEvenTimestep,
    unsigned int *indicesCoarse000,
    unsigned int *indicesFineMMM,
    unsigned int numberOfInterfaceNodes,
    real omegaCoarse,
    real omegaFine,
    OffFC offsetFC)
{
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get the thread index coordinates from threadIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned k_thread = vf::gpu::getNodeIndex();

    //////////////////////////////////////////////////////////////////////////
    //! - Return for non-interface node
    if (k_thread >= numberOfInterfaceNodes)
        return;

    //////////////////////////////////////////////////////////////////////////
    //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on
    //! timestep is based on the esoteric twist algorithm \ref <a
    //! href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017),
    //! DOI:10.3390/computation5020019 ]</b></a>
    //!
    Distributions27 distFine   = vf::gpu::getDistributionReferences27(distributionsFine,   numberOfLBnodesFine,   true);
    Distributions27 distCoarse = vf::gpu::getDistributionReferences27(distributionsCoarse, numberOfLBnodesCoarse, isEvenTimestep);

    ////////////////////////////////////////////////////////////////////////////////
    //! - declare local variables for source nodes
    //!
    real eps_new = c2o1; // ratio of grid resolutions
    real omegaF  = omegaFine;
    real omegaC  = omegaCoarse;

    // zeroth and first order moments at the source nodes
    real drho_PPP, vx1_PPP, vx2_PPP, vx3_PPP;
    real drho_MPP, vx1_MPP, vx2_MPP, vx3_MPP;
    real drho_PMP, vx1_PMP, vx2_PMP, vx3_PMP;
    real drho_MMP, vx1_MMP, vx2_MMP, vx3_MMP;
    real drho_PPM, vx1_PPM, vx2_PPM, vx3_PPM;
    real drho_MPM, vx1_MPM, vx2_MPM, vx3_MPM;
    real drho_PMM, vx1_PMM, vx2_PMM, vx3_PMM;
    real drho_MMM, vx1_MMM, vx2_MMM, vx3_MMM;

    // second order moments at the source nodes
    real kxyFromfcNEQ_PPP, kyzFromfcNEQ_PPP, kxzFromfcNEQ_PPP, kxxMyyFromfcNEQ_PPP, kxxMzzFromfcNEQ_PPP;
    real kxyFromfcNEQ_MPP, kyzFromfcNEQ_MPP, kxzFromfcNEQ_MPP, kxxMyyFromfcNEQ_MPP, kxxMzzFromfcNEQ_MPP;
    real kxyFromfcNEQ_PMP, kyzFromfcNEQ_PMP, kxzFromfcNEQ_PMP, kxxMyyFromfcNEQ_PMP, kxxMzzFromfcNEQ_PMP;
    real kxyFromfcNEQ_MMP, kyzFromfcNEQ_MMP, kxzFromfcNEQ_MMP, kxxMyyFromfcNEQ_MMP, kxxMzzFromfcNEQ_MMP;
    real kxyFromfcNEQ_PPM, kyzFromfcNEQ_PPM, kxzFromfcNEQ_PPM, kxxMyyFromfcNEQ_PPM, kxxMzzFromfcNEQ_PPM;
    real kxyFromfcNEQ_MPM, kyzFromfcNEQ_MPM, kxzFromfcNEQ_MPM, kxxMyyFromfcNEQ_MPM, kxxMzzFromfcNEQ_MPM;
    real kxyFromfcNEQ_PMM, kyzFromfcNEQ_PMM, kxzFromfcNEQ_PMM, kxxMyyFromfcNEQ_PMM, kxxMzzFromfcNEQ_PMM;
    real kxyFromfcNEQ_MMM, kyzFromfcNEQ_MMM, kxzFromfcNEQ_MMM, kxxMyyFromfcNEQ_MMM, kxxMzzFromfcNEQ_MMM;

    //////////////////////////////////////////////////////////////////////////
    //! - Calculate moments for each source node 
    //!
    //////////////////////////////////////////////////////////////////////////
    // source node BSW = MMM
    //////////////////////////////////////////////////////////////////////////
    // index of the base node and its neighbors
    unsigned int k_base_000 = indicesFineMMM[k_thread];
    unsigned int k_base_M00 = neighborXfine [k_base_000];
    unsigned int k_base_0M0 = neighborYfine [k_base_000];
    unsigned int k_base_00M = neighborZfine [k_base_000];
    unsigned int k_base_MM0 = neighborYfine [k_base_M00];
    unsigned int k_base_M0M = neighborZfine [k_base_M00];
    unsigned int k_base_0MM = neighborZfine [k_base_0M0];
    unsigned int k_base_MMM = neighborZfine [k_base_MM0];
    //////////////////////////////////////////////////////////////////////////
    // Set neighbor indices
    unsigned int k_000 = k_base_000;
    unsigned int k_M00 = k_base_M00;
    unsigned int k_0M0 = k_base_0M0;
    unsigned int k_00M = k_base_00M;
    unsigned int k_MM0 = k_base_MM0;
    unsigned int k_M0M = k_base_M0M;
    unsigned int k_0MM = k_base_0MM;
    unsigned int k_MMM = k_base_MMM;

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_MMM, vx1_MMM, vx2_MMM, vx3_MMM,
        kxyFromfcNEQ_MMM, kyzFromfcNEQ_MMM, kxzFromfcNEQ_MMM, kxxMyyFromfcNEQ_MMM, kxxMzzFromfcNEQ_MMM);

    //////////////////////////////////////////////////////////////////////////
    // source node TSW = MMP
    //////////////////////////////////////////////////////////////////////////
    // Set neighbor indices - has to be recalculated for the new source node
    k_000 = k_00M;
    k_M00 = k_M0M;
    k_0M0 = k_0MM;
    k_00M = neighborZfine[k_00M];
    k_MM0 = k_MMM;
    k_M0M = neighborZfine[k_M0M];
    k_0MM = neighborZfine[k_0MM];
    k_MMM = neighborZfine[k_MMM];

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_MMP, vx1_MMP, vx2_MMP, vx3_MMP,
        kxyFromfcNEQ_MMP, kyzFromfcNEQ_MMP, kxzFromfcNEQ_MMP, kxxMyyFromfcNEQ_MMP, kxxMzzFromfcNEQ_MMP);

    //////////////////////////////////////////////////////////////////////////
    // source node TSE = PMP
    //////////////////////////////////////////////////////////////////////////
    // index
    k_000 = k_M00;
    k_M00 = neighborXfine[k_M00];
    k_0M0 = k_MM0;
    k_00M = k_M0M;
    k_MM0 = neighborXfine[k_MM0];
    k_M0M = neighborXfine[k_M0M];
    k_0MM = k_MMM;
    k_MMM = neighborXfine[k_MMM];

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_PMP, vx1_PMP, vx2_PMP, vx3_PMP,
        kxyFromfcNEQ_PMP, kyzFromfcNEQ_PMP, kxzFromfcNEQ_PMP, kxxMyyFromfcNEQ_PMP, kxxMzzFromfcNEQ_PMP);

    //////////////////////////////////////////////////////////////////////////
    // source node BSE = PMM 
    //////////////////////////////////////////////////////////////////////////
    // index
    k_00M = k_000;
    k_M0M = k_M00;
    k_0MM = k_0M0;
    k_MMM = k_MM0;
    k_000 = k_base_M00;
    k_M00 = neighborXfine[k_base_M00];
    k_0M0 = k_base_MM0;
    k_MM0 = neighborXfine[k_base_MM0];

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_PMM, vx1_PMM, vx2_PMM, vx3_PMM,
        kxyFromfcNEQ_PMM, kyzFromfcNEQ_PMM, kxzFromfcNEQ_PMM, kxxMyyFromfcNEQ_PMM, kxxMzzFromfcNEQ_PMM);

    //////////////////////////////////////////////////////////////////////////
    // source node BNW = MPM
    //////////////////////////////////////////////////////////////////////////
    // index of the base node and its neighbors --> indices of all source nodes
    k_base_000 = k_base_0M0;
    k_base_M00 = k_base_MM0;
    k_base_0M0 = neighborYfine[k_base_0M0];
    k_base_00M = k_base_0MM;
    k_base_MM0 = neighborYfine[k_base_MM0];
    k_base_M0M = k_base_MMM;
    k_base_0MM = neighborYfine[k_base_0MM];
    k_base_MMM = neighborYfine[k_base_MMM];
    //////////////////////////////////////////////////////////////////////////
    // index
    k_000 = k_base_000;
    k_M00 = k_base_M00;
    k_0M0 = k_base_0M0;
    k_00M = k_base_00M;
    k_MM0 = k_base_MM0;
    k_M0M = k_base_M0M;
    k_0MM = k_base_0MM;
    k_MMM = k_base_MMM;

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_MPM, vx1_MPM, vx2_MPM, vx3_MPM,
        kxyFromfcNEQ_MPM, kyzFromfcNEQ_MPM, kxzFromfcNEQ_MPM, kxxMyyFromfcNEQ_MPM, kxxMzzFromfcNEQ_MPM);

    //////////////////////////////////////////////////////////////////////////
    // source node TNW = MPP
    //////////////////////////////////////////////////////////////////////////
    // index
    k_000 = k_00M;
    k_M00 = k_M0M;
    k_0M0 = k_0MM;
    k_00M = neighborZfine[k_00M];
    k_MM0 = k_MMM;
    k_M0M = neighborZfine[k_M0M];
    k_0MM = neighborZfine[k_0MM];
    k_MMM = neighborZfine[k_MMM];
    
    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_MPP, vx1_MPP, vx2_MPP, vx3_MPP,
        kxyFromfcNEQ_MPP, kyzFromfcNEQ_MPP, kxzFromfcNEQ_MPP, kxxMyyFromfcNEQ_MPP, kxxMzzFromfcNEQ_MPP);

    //////////////////////////////////////////////////////////////////////////
    // source node TNE = PPP
    //////////////////////////////////////////////////////////////////////////
    // index
    k_000 = k_M00;
    k_M00 = neighborXfine[k_M00];
    k_0M0 = k_MM0;
    k_00M = k_M0M;
    k_MM0 = neighborXfine[k_MM0];
    k_M0M = neighborXfine[k_M0M];
    k_0MM = k_MMM;
    k_MMM = neighborXfine[k_MMM];

    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_PPP, vx1_PPP, vx2_PPP, vx3_PPP,
        kxyFromfcNEQ_PPP, kyzFromfcNEQ_PPP, kxzFromfcNEQ_PPP, kxxMyyFromfcNEQ_PPP, kxxMzzFromfcNEQ_PPP);

    //////////////////////////////////////////////////////////////////////////
    // source node BNE = PPM
    //////////////////////////////////////////////////////////////////////////
    // index
    k_00M = k_000;
    k_M0M = k_M00;
    k_0MM = k_0M0;
    k_MMM = k_MM0;
    k_000 = k_base_M00;
    k_M00 = neighborXfine[k_base_M00];
    k_0M0 = k_base_MM0;
    k_MM0 = neighborXfine[k_base_MM0];
    
    calculateMomentsOnSourceNodes( distFine, omegaF,
        k_000, k_M00, k_0M0, k_00M, k_MM0, k_M0M, k_0MM, k_MMM, drho_PPM, vx1_PPM, vx2_PPM, vx3_PPM,
        kxyFromfcNEQ_PPM, kyzFromfcNEQ_PPM, kxzFromfcNEQ_PPM, kxxMyyFromfcNEQ_PPM, kxxMzzFromfcNEQ_PPM);

    //////////////////////////////////////////////////////////////////////////
    //! - Calculate coefficients for polynomial interpolation
    //!
    // example: a_110: derivation in x and y direction
    real a_000, a_100, a_010, a_001, a_200, a_020, a_002, a_110, a_101, a_011;
    real b_000, b_100, b_010, b_001, b_200, b_020, b_002, b_110, b_101, b_011;
    real c_000, c_100, c_010, c_001, c_200, c_020, c_002, c_110, c_101, c_011;
    real d_000, d_100, d_010, d_001, d_110, d_101, d_011;

    a_000 = (-kxxMyyFromfcNEQ_PPM - kxxMyyFromfcNEQ_PPP + kxxMyyFromfcNEQ_MPM + kxxMyyFromfcNEQ_MPP -
            kxxMyyFromfcNEQ_PMM - kxxMyyFromfcNEQ_PMP + kxxMyyFromfcNEQ_MMM + kxxMyyFromfcNEQ_MMP -
            kxxMzzFromfcNEQ_PPM - kxxMzzFromfcNEQ_PPP + kxxMzzFromfcNEQ_MPM + kxxMzzFromfcNEQ_MPP -
            kxxMzzFromfcNEQ_PMM - kxxMzzFromfcNEQ_PMP + kxxMzzFromfcNEQ_MMM + kxxMzzFromfcNEQ_MMP -
            c2o1 * kxyFromfcNEQ_PPM - c2o1 * kxyFromfcNEQ_PPP - c2o1 * kxyFromfcNEQ_MPM - c2o1 * kxyFromfcNEQ_MPP +
            c2o1 * kxyFromfcNEQ_PMM + c2o1 * kxyFromfcNEQ_PMP + c2o1 * kxyFromfcNEQ_MMM + c2o1 * kxyFromfcNEQ_MMP +
            c2o1 * kxzFromfcNEQ_PPM - c2o1 * kxzFromfcNEQ_PPP + c2o1 * kxzFromfcNEQ_MPM - c2o1 * kxzFromfcNEQ_MPP +
            c2o1 * kxzFromfcNEQ_PMM - c2o1 * kxzFromfcNEQ_PMP + c2o1 * kxzFromfcNEQ_MMM - c2o1 * kxzFromfcNEQ_MMP +
            c8o1 * vx1_PPM + c8o1 * vx1_PPP + c8o1 * vx1_MPM + c8o1 * vx1_MPP + c8o1 * vx1_PMM + c8o1 * vx1_PMP +
            c8o1 * vx1_MMM + c8o1 * vx1_MMP + c2o1 * vx2_PPM + c2o1 * vx2_PPP - c2o1 * vx2_MPM - c2o1 * vx2_MPP -
            c2o1 * vx2_PMM - c2o1 * vx2_PMP + c2o1 * vx2_MMM + c2o1 * vx2_MMP - c2o1 * vx3_PPM + c2o1 * vx3_PPP +
            c2o1 * vx3_MPM - c2o1 * vx3_MPP - c2o1 * vx3_PMM + c2o1 * vx3_PMP + c2o1 * vx3_MMM - c2o1 * vx3_MMP) /
            c64o1;
    b_000 = (c2o1 * kxxMyyFromfcNEQ_PPM + c2o1 * kxxMyyFromfcNEQ_PPP + c2o1 * kxxMyyFromfcNEQ_MPM +
            c2o1 * kxxMyyFromfcNEQ_MPP - c2o1 * kxxMyyFromfcNEQ_PMM - c2o1 * kxxMyyFromfcNEQ_PMP -
            c2o1 * kxxMyyFromfcNEQ_MMM - c2o1 * kxxMyyFromfcNEQ_MMP - kxxMzzFromfcNEQ_PPM - kxxMzzFromfcNEQ_PPP -
            kxxMzzFromfcNEQ_MPM - kxxMzzFromfcNEQ_MPP + kxxMzzFromfcNEQ_PMM + kxxMzzFromfcNEQ_PMP +
            kxxMzzFromfcNEQ_MMM + kxxMzzFromfcNEQ_MMP - c2o1 * kxyFromfcNEQ_PPM - c2o1 * kxyFromfcNEQ_PPP +
            c2o1 * kxyFromfcNEQ_MPM + c2o1 * kxyFromfcNEQ_MPP - c2o1 * kxyFromfcNEQ_PMM - c2o1 * kxyFromfcNEQ_PMP +
            c2o1 * kxyFromfcNEQ_MMM + c2o1 * kxyFromfcNEQ_MMP + c2o1 * kyzFromfcNEQ_PPM - c2o1 * kyzFromfcNEQ_PPP +
            c2o1 * kyzFromfcNEQ_MPM - c2o1 * kyzFromfcNEQ_MPP + c2o1 * kyzFromfcNEQ_PMM - c2o1 * kyzFromfcNEQ_PMP +
            c2o1 * kyzFromfcNEQ_MMM - c2o1 * kyzFromfcNEQ_MMP + c2o1 * vx1_PPM + c2o1 * vx1_PPP - c2o1 * vx1_MPM -
            c2o1 * vx1_MPP - c2o1 * vx1_PMM - c2o1 * vx1_PMP + c2o1 * vx1_MMM + c2o1 * vx1_MMP + c8o1 * vx2_PPM +
            c8o1 * vx2_PPP + c8o1 * vx2_MPM + c8o1 * vx2_MPP + c8o1 * vx2_PMM + c8o1 * vx2_PMP + c8o1 * vx2_MMM +
            c8o1 * vx2_MMP - c2o1 * vx3_PPM + c2o1 * vx3_PPP - c2o1 * vx3_MPM + c2o1 * vx3_MPP + c2o1 * vx3_PMM -
            c2o1 * vx3_PMP + c2o1 * vx3_MMM - c2o1 * vx3_MMP) /
            c64o1;
    c_000 = (kxxMyyFromfcNEQ_PPM - kxxMyyFromfcNEQ_PPP + kxxMyyFromfcNEQ_MPM - kxxMyyFromfcNEQ_MPP +
            kxxMyyFromfcNEQ_PMM - kxxMyyFromfcNEQ_PMP + kxxMyyFromfcNEQ_MMM - kxxMyyFromfcNEQ_MMP -
            c2o1 * kxxMzzFromfcNEQ_PPM + c2o1 * kxxMzzFromfcNEQ_PPP - c2o1 * kxxMzzFromfcNEQ_MPM +
            c2o1 * kxxMzzFromfcNEQ_MPP - c2o1 * kxxMzzFromfcNEQ_PMM + c2o1 * kxxMzzFromfcNEQ_PMP -
            c2o1 * kxxMzzFromfcNEQ_MMM + c2o1 * kxxMzzFromfcNEQ_MMP - c2o1 * kxzFromfcNEQ_PPM -
            c2o1 * kxzFromfcNEQ_PPP + c2o1 * kxzFromfcNEQ_MPM + c2o1 * kxzFromfcNEQ_MPP - c2o1 * kxzFromfcNEQ_PMM -
            c2o1 * kxzFromfcNEQ_PMP + c2o1 * kxzFromfcNEQ_MMM + c2o1 * kxzFromfcNEQ_MMP - c2o1 * kyzFromfcNEQ_PPM -
            c2o1 * kyzFromfcNEQ_PPP - c2o1 * kyzFromfcNEQ_MPM - c2o1 * kyzFromfcNEQ_MPP + c2o1 * kyzFromfcNEQ_PMM +
            c2o1 * kyzFromfcNEQ_PMP + c2o1 * kyzFromfcNEQ_MMM + c2o1 * kyzFromfcNEQ_MMP - c2o1 * vx1_PPM +
            c2o1 * vx1_PPP + c2o1 * vx1_MPM - c2o1 * vx1_MPP - c2o1 * vx1_PMM + c2o1 * vx1_PMP + c2o1 * vx1_MMM -
            c2o1 * vx1_MMP - c2o1 * vx2_PPM + c2o1 * vx2_PPP - c2o1 * vx2_MPM + c2o1 * vx2_MPP + c2o1 * vx2_PMM -
            c2o1 * vx2_PMP + c2o1 * vx2_MMM - c2o1 * vx2_MMP + c8o1 * vx3_PPM + c8o1 * vx3_PPP + c8o1 * vx3_MPM +
            c8o1 * vx3_MPP + c8o1 * vx3_PMM + c8o1 * vx3_PMP + c8o1 * vx3_MMM + c8o1 * vx3_MMP) /
            c64o1;
    a_100  = (vx1_PPM + vx1_PPP - vx1_MPM - vx1_MPP + vx1_PMM + vx1_PMP - vx1_MMM - vx1_MMP) / c4o1;
    b_100  = (vx2_PPM + vx2_PPP - vx2_MPM - vx2_MPP + vx2_PMM + vx2_PMP - vx2_MMM - vx2_MMP) / c4o1;
    c_100  = (vx3_PPM + vx3_PPP - vx3_MPM - vx3_MPP + vx3_PMM + vx3_PMP - vx3_MMM - vx3_MMP) / c4o1;
    a_200 = (kxxMyyFromfcNEQ_PPM + kxxMyyFromfcNEQ_PPP - kxxMyyFromfcNEQ_MPM - kxxMyyFromfcNEQ_MPP +
            kxxMyyFromfcNEQ_PMM + kxxMyyFromfcNEQ_PMP - kxxMyyFromfcNEQ_MMM - kxxMyyFromfcNEQ_MMP +
            kxxMzzFromfcNEQ_PPM + kxxMzzFromfcNEQ_PPP - kxxMzzFromfcNEQ_MPM - kxxMzzFromfcNEQ_MPP +
            kxxMzzFromfcNEQ_PMM + kxxMzzFromfcNEQ_PMP - kxxMzzFromfcNEQ_MMM - kxxMzzFromfcNEQ_MMP + c2o1 * vx2_PPM +
            c2o1 * vx2_PPP - c2o1 * vx2_MPM - c2o1 * vx2_MPP - c2o1 * vx2_PMM - c2o1 * vx2_PMP + c2o1 * vx2_MMM +
            c2o1 * vx2_MMP - c2o1 * vx3_PPM + c2o1 * vx3_PPP + c2o1 * vx3_MPM - c2o1 * vx3_MPP - c2o1 * vx3_PMM +
            c2o1 * vx3_PMP + c2o1 * vx3_MMM - c2o1 * vx3_MMP) /
            c16o1;
    b_200 = (kxyFromfcNEQ_PPM + kxyFromfcNEQ_PPP - kxyFromfcNEQ_MPM - kxyFromfcNEQ_MPP + kxyFromfcNEQ_PMM +
            kxyFromfcNEQ_PMP - kxyFromfcNEQ_MMM - kxyFromfcNEQ_MMP - c2o1 * vx1_PPM - c2o1 * vx1_PPP +
            c2o1 * vx1_MPM + c2o1 * vx1_MPP + c2o1 * vx1_PMM + c2o1 * vx1_PMP - c2o1 * vx1_MMM - c2o1 * vx1_MMP) /
            c8o1;
    c_200 = (kxzFromfcNEQ_PPM + kxzFromfcNEQ_PPP - kxzFromfcNEQ_MPM - kxzFromfcNEQ_MPP + kxzFromfcNEQ_PMM +
            kxzFromfcNEQ_PMP - kxzFromfcNEQ_MMM - kxzFromfcNEQ_MMP + c2o1 * vx1_PPM - c2o1 * vx1_PPP -
            c2o1 * vx1_MPM + c2o1 * vx1_MPP + c2o1 * vx1_PMM - c2o1 * vx1_PMP - c2o1 * vx1_MMM + c2o1 * vx1_MMP) /
            c8o1;
    a_010  = (vx1_PPM + vx1_PPP + vx1_MPM + vx1_MPP - vx1_PMM - vx1_PMP - vx1_MMM - vx1_MMP) / c4o1;
    b_010  = (vx2_PPM + vx2_PPP + vx2_MPM + vx2_MPP - vx2_PMM - vx2_PMP - vx2_MMM - vx2_MMP) / c4o1;
    c_010  = (vx3_PPM + vx3_PPP + vx3_MPM + vx3_MPP - vx3_PMM - vx3_PMP - vx3_MMM - vx3_MMP) / c4o1;
    a_020 = (kxyFromfcNEQ_PPM + kxyFromfcNEQ_PPP + kxyFromfcNEQ_MPM + kxyFromfcNEQ_MPP - kxyFromfcNEQ_PMM -
            kxyFromfcNEQ_PMP - kxyFromfcNEQ_MMM - kxyFromfcNEQ_MMP - c2o1 * vx2_PPM - c2o1 * vx2_PPP +
            c2o1 * vx2_MPM + c2o1 * vx2_MPP + c2o1 * vx2_PMM + c2o1 * vx2_PMP - c2o1 * vx2_MMM - c2o1 * vx2_MMP) /
            c8o1;
    b_020 = (-c2o1 * kxxMyyFromfcNEQ_PPM - c2o1 * kxxMyyFromfcNEQ_PPP - c2o1 * kxxMyyFromfcNEQ_MPM -
            c2o1 * kxxMyyFromfcNEQ_MPP + c2o1 * kxxMyyFromfcNEQ_PMM + c2o1 * kxxMyyFromfcNEQ_PMP +
            c2o1 * kxxMyyFromfcNEQ_MMM + c2o1 * kxxMyyFromfcNEQ_MMP + kxxMzzFromfcNEQ_PPM + kxxMzzFromfcNEQ_PPP +
            kxxMzzFromfcNEQ_MPM + kxxMzzFromfcNEQ_MPP - kxxMzzFromfcNEQ_PMM - kxxMzzFromfcNEQ_PMP -
            kxxMzzFromfcNEQ_MMM - kxxMzzFromfcNEQ_MMP + c2o1 * vx1_PPM + c2o1 * vx1_PPP - c2o1 * vx1_MPM -
            c2o1 * vx1_MPP - c2o1 * vx1_PMM - c2o1 * vx1_PMP + c2o1 * vx1_MMM + c2o1 * vx1_MMP - c2o1 * vx3_PPM +
            c2o1 * vx3_PPP - c2o1 * vx3_MPM + c2o1 * vx3_MPP + c2o1 * vx3_PMM - c2o1 * vx3_PMP + c2o1 * vx3_MMM -
            c2o1 * vx3_MMP) /
            c16o1;
    c_020 = (kyzFromfcNEQ_PPM + kyzFromfcNEQ_PPP + kyzFromfcNEQ_MPM + kyzFromfcNEQ_MPP - kyzFromfcNEQ_PMM -
            kyzFromfcNEQ_PMP - kyzFromfcNEQ_MMM - kyzFromfcNEQ_MMP + c2o1 * vx2_PPM - c2o1 * vx2_PPP +
            c2o1 * vx2_MPM - c2o1 * vx2_MPP - c2o1 * vx2_PMM + c2o1 * vx2_PMP - c2o1 * vx2_MMM + c2o1 * vx2_MMP) /
            c8o1;
    a_001  = (-vx1_PPM + vx1_PPP - vx1_MPM + vx1_MPP - vx1_PMM + vx1_PMP - vx1_MMM + vx1_MMP) / c4o1;
    b_001  = (-vx2_PPM + vx2_PPP - vx2_MPM + vx2_MPP - vx2_PMM + vx2_PMP - vx2_MMM + vx2_MMP) / c4o1;
    c_001  = (-vx3_PPM + vx3_PPP - vx3_MPM + vx3_MPP - vx3_PMM + vx3_PMP - vx3_MMM + vx3_MMP) / c4o1;
    a_002 = (-kxzFromfcNEQ_PPM + kxzFromfcNEQ_PPP - kxzFromfcNEQ_MPM + kxzFromfcNEQ_MPP - kxzFromfcNEQ_PMM +
            kxzFromfcNEQ_PMP - kxzFromfcNEQ_MMM + kxzFromfcNEQ_MMP + c2o1 * vx3_PPM - c2o1 * vx3_PPP -
            c2o1 * vx3_MPM + c2o1 * vx3_MPP + c2o1 * vx3_PMM - c2o1 * vx3_PMP - c2o1 * vx3_MMM + c2o1 * vx3_MMP) /
            c8o1;
    b_002 = (-kyzFromfcNEQ_PPM + kyzFromfcNEQ_PPP - kyzFromfcNEQ_MPM + kyzFromfcNEQ_MPP - kyzFromfcNEQ_PMM +
            kyzFromfcNEQ_PMP - kyzFromfcNEQ_MMM + kyzFromfcNEQ_MMP + c2o1 * vx3_PPM - c2o1 * vx3_PPP +
            c2o1 * vx3_MPM - c2o1 * vx3_MPP - c2o1 * vx3_PMM + c2o1 * vx3_PMP - c2o1 * vx3_MMM + c2o1 * vx3_MMP) /
            c8o1;
    c_002 = (-kxxMyyFromfcNEQ_PPM + kxxMyyFromfcNEQ_PPP - kxxMyyFromfcNEQ_MPM + kxxMyyFromfcNEQ_MPP -
            kxxMyyFromfcNEQ_PMM + kxxMyyFromfcNEQ_PMP - kxxMyyFromfcNEQ_MMM + kxxMyyFromfcNEQ_MMP +
            c2o1 * kxxMzzFromfcNEQ_PPM - c2o1 * kxxMzzFromfcNEQ_PPP + c2o1 * kxxMzzFromfcNEQ_MPM -
            c2o1 * kxxMzzFromfcNEQ_MPP + c2o1 * kxxMzzFromfcNEQ_PMM - c2o1 * kxxMzzFromfcNEQ_PMP +
            c2o1 * kxxMzzFromfcNEQ_MMM - c2o1 * kxxMzzFromfcNEQ_MMP - c2o1 * vx1_PPM + c2o1 * vx1_PPP +
            c2o1 * vx1_MPM - c2o1 * vx1_MPP - c2o1 * vx1_PMM + c2o1 * vx1_PMP + c2o1 * vx1_MMM - c2o1 * vx1_MMP -
            c2o1 * vx2_PPM + c2o1 * vx2_PPP - c2o1 * vx2_MPM + c2o1 * vx2_MPP + c2o1 * vx2_PMM - c2o1 * vx2_PMP +
            c2o1 * vx2_MMM - c2o1 * vx2_MMP) /
            c16o1;
    a_110 = (vx1_PPM + vx1_PPP - vx1_MPM - vx1_MPP - vx1_PMM - vx1_PMP + vx1_MMM + vx1_MMP) / c2o1;
    b_110 = (vx2_PPM + vx2_PPP - vx2_MPM - vx2_MPP - vx2_PMM - vx2_PMP + vx2_MMM + vx2_MMP) / c2o1;
    c_110 = (vx3_PPM + vx3_PPP - vx3_MPM - vx3_MPP - vx3_PMM - vx3_PMP + vx3_MMM + vx3_MMP) / c2o1;
    a_101 = (-vx1_PPM + vx1_PPP + vx1_MPM - vx1_MPP - vx1_PMM + vx1_PMP + vx1_MMM - vx1_MMP) / c2o1;
    b_101 = (-vx2_PPM + vx2_PPP + vx2_MPM - vx2_MPP - vx2_PMM + vx2_PMP + vx2_MMM - vx2_MMP) / c2o1;
    c_101 = (-vx3_PPM + vx3_PPP + vx3_MPM - vx3_MPP - vx3_PMM + vx3_PMP + vx3_MMM - vx3_MMP) / c2o1;
    a_011 = (-vx1_PPM + vx1_PPP - vx1_MPM + vx1_MPP + vx1_PMM - vx1_PMP + vx1_MMM - vx1_MMP) / c2o1;
    b_011 = (-vx2_PPM + vx2_PPP - vx2_MPM + vx2_MPP + vx2_PMM - vx2_PMP + vx2_MMM - vx2_MMP) / c2o1;
    c_011 = (-vx3_PPM + vx3_PPP - vx3_MPM + vx3_MPP + vx3_PMM - vx3_PMP + vx3_MMM - vx3_MMP) / c2o1;

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    
    real kxyAverage    = c0o1;
    real kyzAverage    = c0o1;
    real kxzAverage    = c0o1;
    real kxxMyyAverage = c0o1;
    real kxxMzzAverage = c0o1;

    ////////////////////////////////////////////////////////////////////////////////
    //! - Set the relative position of the offset cell {-1, 0, 1}
    //!
    real xoff    = offsetFC.xOffFC[k_thread];
    real yoff    = offsetFC.yOffFC[k_thread];
    real zoff    = offsetFC.zOffFC[k_thread];
     
    real xoff_sq = xoff * xoff;
    real yoff_sq = yoff * yoff;
    real zoff_sq = zoff * zoff;

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //!- Calculate coefficients for the polynomial interpolation of the pressure
    //! 
    real LaplaceRho = 
        ((xoff != c0o1) || (yoff != c0o1) || (zoff != c0o1))
        ? c0o1 : c0o1;
//        : -c3o1 * (a_100 * a_100 + b_010 * b_010 + c_001 * c_001) - c6o1 * (b_100 * a_010 + c_100 * a_001 + c_010 * b_001);
    d_000 = ( drho_PPM + drho_PPP + drho_MPM + drho_MPP + drho_PMM + drho_PMP + drho_MMM + drho_MMP - c2o1 * LaplaceRho) * c1o8;
    d_100 = ( drho_PPM + drho_PPP - drho_MPM - drho_MPP + drho_PMM + drho_PMP - drho_MMM - drho_MMP) * c1o4;
    d_010 = ( drho_PPM + drho_PPP + drho_MPM + drho_MPP - drho_PMM - drho_PMP - drho_MMM - drho_MMP) * c1o4;
    d_001 = (-drho_PPM + drho_PPP - drho_MPM + drho_MPP - drho_PMM + drho_PMP - drho_MMM + drho_MMP) * c1o4;
    d_110 = ( drho_PPM + drho_PPP - drho_MPM - drho_MPP - drho_PMM - drho_PMP + drho_MMM + drho_MMP) * c1o2;
    d_101 = (-drho_PPM + drho_PPP + drho_MPM - drho_MPP - drho_PMM + drho_PMP + drho_MMM - drho_MMP) * c1o2;
    d_011 = (-drho_PPM + drho_PPP - drho_MPM + drho_MPP + drho_PMM - drho_PMP + drho_MMM - drho_MMP) * c1o2;


    //////////////////////////////////////////////////////////////////////////
    //! - Extrapolation for refinement in to the wall (polynomial coefficients)
    //!
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //
    // x------x
    // |      |
    // |   ---+--->X
    // |      |  \
    // x------x   \
    //          offset-vector
    //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    a_000 = a_000 + xoff * a_100 + yoff * a_010 + zoff * a_001 + xoff_sq * a_200 + yoff_sq * a_020 + zoff_sq * a_002 +
            xoff * yoff * a_110 + xoff * zoff * a_101 + yoff * zoff * a_011;
    a_100 = a_100 + c2o1 * xoff * a_200 + yoff * a_110 + zoff * a_101;
    a_010 = a_010 + c2o1 * yoff * a_020 + xoff * a_110 + zoff * a_011;
    a_001 = a_001 + c2o1 * zoff * a_002 + xoff * a_101 + yoff * a_011;
    b_000 = b_000 + xoff * b_100 + yoff * b_010 + zoff * b_001 + xoff_sq * b_200 + yoff_sq * b_020 + zoff_sq * b_002 +
            xoff * yoff * b_110 + xoff * zoff * b_101 + yoff * zoff * b_011;
    b_100 = b_100 + c2o1 * xoff * b_200 + yoff * b_110 + zoff * b_101;
    b_010 = b_010 + c2o1 * yoff * b_020 + xoff * b_110 + zoff * b_011;
    b_001 = b_001 + c2o1 * zoff * b_002 + xoff * b_101 + yoff * b_011;
    c_000 = c_000 + xoff * c_100 + yoff * c_010 + zoff * c_001 + xoff_sq * c_200 + yoff_sq * c_020 + zoff_sq * c_002 +
            xoff * yoff * c_110 + xoff * zoff * c_101 + yoff * zoff * c_011;
    c_100 = c_100 + c2o1 * xoff * c_200 + yoff * c_110 + zoff * c_101;
    c_010 = c_010 + c2o1 * yoff * c_020 + xoff * c_110 + zoff * c_011;
    c_001 = c_001 + c2o1 * zoff * c_002 + xoff * c_101 + yoff * c_011;
    d_000 = d_000 + xoff * d_100 + yoff * d_010 + zoff * d_001 + 
            xoff * yoff * d_110 + xoff * zoff * d_101 + yoff * zoff * d_011;

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Set all moments to zero
    //!
    real m_111 = c0o1;
    real m_211 = c0o1;
    real m_011 = c0o1;
    real m_121 = c0o1;
    real m_101 = c0o1;
    real m_112 = c0o1;
    real m_110 = c0o1;
    real m_221 = c0o1;
    real m_001 = c0o1;
    real m_201 = c0o1;
    real m_021 = c0o1;
    real m_212 = c0o1;
    real m_010 = c0o1;
    real m_210 = c0o1;
    real m_012 = c0o1;
    real m_122 = c0o1;
    real m_100 = c0o1;
    real m_120 = c0o1;
    real m_102 = c0o1;
    real m_222 = c0o1;
    real m_022 = c0o1;
    real m_202 = c0o1;
    real m_002 = c0o1;
    real m_220 = c0o1;
    real m_020 = c0o1;
    real m_200 = c0o1;
    real m_000 = c0o1;

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Define aliases to use the same variable for the distributions (f's):
    //!
    real& f_000 = m_111;
    real& f_P00 = m_211;
    real& f_M00 = m_011;
    real& f_0P0 = m_121;
    real& f_0M0 = m_101;
    real& f_00P = m_112;
    real& f_00M = m_110;
    real& f_PP0 = m_221;
    real& f_MM0 = m_001;
    real& f_PM0 = m_201;
    real& f_MP0 = m_021;
    real& f_P0P = m_212;
    real& f_M0M = m_010;
    real& f_P0M = m_210;
    real& f_M0P = m_012;
    real& f_0PP = m_122;
    real& f_0MM = m_100;
    real& f_0PM = m_120;
    real& f_0MP = m_102;
    real& f_PPP = m_222;
    real& f_MPP = m_022;
    real& f_PMP = m_202;
    real& f_MMP = m_002;
    real& f_PPM = m_220;
    real& f_MPM = m_020;
    real& f_PMM = m_200;
    real& f_MMM = m_000;

    ////////////////////////////////////////////////////////////////////////////////
    //! - Declare local variables for destination nodes
    //!
    real vvx, vvy, vvz, vx_sq, vy_sq, vz_sq;
    real mxxPyyPzz, mxxMyy, mxxMzz, mxxyPyzz, mxxyMyzz, mxxzPyyz, mxxzMyyz, mxyyPxzz, mxyyMxzz;
    real useNEQ = c1o1; // zero; //one;   //.... one = on ..... zero = off
    real press;
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //
    // Position Coarse 0., 0., 0.
    //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // x = 0.;
    // y = 0.;
    // z = 0.;
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////////////////////////
    //! - Set macroscopic values on destination node (zeroth and first order moments)
    //!
    press = d_000;
    vvx   = a_000;
    vvy   = b_000;
    vvz   = c_000;

    m_000 = press; // m_000 is press, if drho is interpolated directly

    vx_sq = vvx * vvx;
    vy_sq = vvy * vvy;
    vz_sq = vvz * vvz;

    ////////////////////////////////////////////////////////////////////////////////
    //! - Set moments (second to sixth order) on destination node
    //!
    // linear combinations for second order moments
    mxxPyyPzz = m_000;

    mxxMyy = -c2o3 * ((a_100 - b_010) + kxxMyyAverage) * eps_new / omegaC * (c1o1 + press);
    mxxMzz = -c2o3 * ((a_100 - c_001) + kxxMzzAverage) * eps_new / omegaC * (c1o1 + press);

    m_011 = -c1o3 * ((b_001 + c_010) + kyzAverage) * eps_new / omegaC * (c1o1 + press);
    m_101 = -c1o3 * ((a_001 + c_100) + kxzAverage) * eps_new / omegaC * (c1o1 + press);
    m_110 = -c1o3 * ((a_010 + b_100) + kxyAverage) * eps_new / omegaC * (c1o1 + press);

    m_200 = c1o3 * (        mxxMyy +        mxxMzz + mxxPyyPzz) * useNEQ;
    m_020 = c1o3 * (-c2o1 * mxxMyy +        mxxMzz + mxxPyyPzz) * useNEQ;
    m_002 = c1o3 * (        mxxMyy - c2o1 * mxxMzz + mxxPyyPzz) * useNEQ;

    // linear combinations for third order moments
    m_111 = c0o1;

    mxxyPyzz = c0o1;
    mxxyMyzz = c0o1;
    mxxzPyyz = c0o1;
    mxxzMyyz = c0o1;
    mxyyPxzz = c0o1;
    mxyyMxzz = c0o1;

    m_210 = ( mxxyMyzz + mxxyPyzz) * c1o2;
    m_012 = (-mxxyMyzz + mxxyPyzz) * c1o2;
    m_201 = ( mxxzMyyz + mxxzPyyz) * c1o2;
    m_021 = (-mxxzMyyz + mxxzPyyz) * c1o2;
    m_120 = ( mxyyMxzz + mxyyPxzz) * c1o2;
    m_102 = (-mxyyMxzz + mxyyPxzz) * c1o2;

    // fourth order moments
    m_022 = m_000 * c1o9;
    m_202 = m_022;
    m_220 = m_022;

    // fifth order moments

    // sixth order moments
    m_222 = m_000 * c1o27;

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Chimera transform from central moments to well conditioned distributions as defined in Appendix J in
    //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015),
    //! DOI:10.1016/j.camwa.2015.05.001 ]</b></a> see also Eq. (88)-(96) in <a
    //! href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040
    //! ]</b></a>
    //!
    ////////////////////////////////////////////////////////////////////////////////////
    // X - Dir
    backwardInverseChimeraWithK(m_000, m_100, m_200, vvx, vx_sq, c1o1, c1o1);
    backwardChimera(            m_010, m_110, m_210, vvx, vx_sq);
    backwardInverseChimeraWithK(m_020, m_120, m_220, vvx, vx_sq, c3o1, c1o3);
    backwardChimera(            m_001, m_101, m_201, vvx, vx_sq);
    backwardChimera(            m_011, m_111, m_211, vvx, vx_sq);
    backwardChimera(            m_021, m_121, m_221, vvx, vx_sq);
    backwardInverseChimeraWithK(m_002, m_102, m_202, vvx, vx_sq, c3o1, c1o3);
    backwardChimera(            m_012, m_112, m_212, vvx, vx_sq);
    backwardInverseChimeraWithK(m_022, m_122, m_222, vvx, vx_sq, c9o1, c1o9);

    ////////////////////////////////////////////////////////////////////////////////////
    // Y - Dir
    backwardInverseChimeraWithK(m_000, m_010, m_020, vvy, vy_sq, c6o1, c1o6);
    backwardChimera(            m_001, m_011, m_021, vvy, vy_sq);
    backwardInverseChimeraWithK(m_002, m_012, m_022, vvy, vy_sq, c18o1, c1o18);
    backwardInverseChimeraWithK(m_100, m_110, m_120, vvy, vy_sq, c3o2, c2o3);
    backwardChimera(            m_101, m_111, m_121, vvy, vy_sq);
    backwardInverseChimeraWithK(m_102, m_112, m_122, vvy, vy_sq, c9o2, c2o9);
    backwardInverseChimeraWithK(m_200, m_210, m_220, vvy, vy_sq, c6o1, c1o6);
    backwardChimera(            m_201, m_211, m_221, vvy, vy_sq);
    backwardInverseChimeraWithK(m_202, m_212, m_222, vvy, vy_sq, c18o1, c1o18);

    ////////////////////////////////////////////////////////////////////////////////////
    // Z - Dir
    backwardInverseChimeraWithK(m_000, m_001, m_002, vvz, vz_sq, c36o1, c1o36);
    backwardInverseChimeraWithK(m_010, m_011, m_012, vvz, vz_sq, c9o1,  c1o9);
    backwardInverseChimeraWithK(m_020, m_021, m_022, vvz, vz_sq, c36o1, c1o36);
    backwardInverseChimeraWithK(m_100, m_101, m_102, vvz, vz_sq, c9o1,  c1o9);
    backwardInverseChimeraWithK(m_110, m_111, m_112, vvz, vz_sq, c9o4,  c4o9);
    backwardInverseChimeraWithK(m_120, m_121, m_122, vvz, vz_sq, c9o1,  c1o9);
    backwardInverseChimeraWithK(m_200, m_201, m_202, vvz, vz_sq, c36o1, c1o36);
    backwardInverseChimeraWithK(m_210, m_211, m_212, vvz, vz_sq, c9o1,  c1o9);
    backwardInverseChimeraWithK(m_220, m_221, m_222, vvz, vz_sq, c36o1, c1o36);


    ////////////////////////////////////////////////////////////////////////////////////
    // index of the destination node and its neighbors
    k_000 = indicesCoarse000[k_thread];
    k_M00 = neighborXcoarse [k_000];
    k_0M0 = neighborYcoarse [k_000];
    k_00M = neighborZcoarse [k_000];
    k_MM0 = neighborYcoarse [k_M00];
    k_M0M = neighborZcoarse [k_M00];
    k_0MM = neighborZcoarse [k_0M0];
    k_MMM = neighborZcoarse [k_MM0];
    ////////////////////////////////////////////////////////////////////////////////////

    ////////////////////////////////////////////////////////////////////////////////////
    //! - Write distributions: style of reading and writing the distributions from/to
    //! stored arrays dependent on timestep is based on the esoteric twist algorithm
    //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017),
    //! DOI:10.3390/computation5020019 ]</b></a>
    //!
    (distCoarse.f[DIR_000])[k_000] = f_000;
    (distCoarse.f[DIR_P00])[k_000] = f_P00;
    (distCoarse.f[DIR_M00])[k_M00] = f_M00;
    (distCoarse.f[DIR_0P0])[k_000] = f_0P0;
    (distCoarse.f[DIR_0M0])[k_0M0] = f_0M0;
    (distCoarse.f[DIR_00P])[k_000] = f_00P;
    (distCoarse.f[DIR_00M])[k_00M] = f_00M;
    (distCoarse.f[DIR_PP0])[k_000] = f_PP0;
    (distCoarse.f[DIR_MM0])[k_MM0] = f_MM0;
    (distCoarse.f[DIR_PM0])[k_0M0] = f_PM0;
    (distCoarse.f[DIR_MP0])[k_M00] = f_MP0;
    (distCoarse.f[DIR_P0P])[k_000] = f_P0P;
    (distCoarse.f[DIR_M0M])[k_M0M] = f_M0M;
    (distCoarse.f[DIR_P0M])[k_00M] = f_P0M;
    (distCoarse.f[DIR_M0P])[k_M00] = f_M0P;
    (distCoarse.f[DIR_0PP])[k_000] = f_0PP;
    (distCoarse.f[DIR_0MM])[k_0MM] = f_0MM;
    (distCoarse.f[DIR_0PM])[k_00M] = f_0PM;
    (distCoarse.f[DIR_0MP])[k_0M0] = f_0MP;
    (distCoarse.f[DIR_PPP])[k_000] = f_PPP;
    (distCoarse.f[DIR_MPP])[k_M00] = f_MPP;
    (distCoarse.f[DIR_PMP])[k_0M0] = f_PMP;
    (distCoarse.f[DIR_MMP])[k_MM0] = f_MMP;
    (distCoarse.f[DIR_PPM])[k_00M] = f_PPM;
    (distCoarse.f[DIR_MPM])[k_M0M] = f_MPM;
    (distCoarse.f[DIR_PMM])[k_0MM] = f_PMM;
    (distCoarse.f[DIR_MMM])[k_MMM] = f_MMM;
    ////////////////////////////////////////////////////////////////////////////////////
}
