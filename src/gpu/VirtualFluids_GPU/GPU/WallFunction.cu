#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;


//////////////////////////////////////////////////////////////////////////////
__global__ void WallFunction27(
										  real* vx,
										  real* vy,
										  real* vz,
										  real* DD, 
										  int* k_Q, 
										  real* QQ,
										  unsigned int numberOfBCnodes, 
										  real om1, 
										  unsigned int* neighborX,
										  unsigned int* neighborY,
										  unsigned int* neighborZ,
										  unsigned long long numberOfLBnodes, 
										  bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[DIR_P00] = &DD[DIR_P00 * numberOfLBnodes];
      D.f[DIR_M00] = &DD[DIR_M00 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
   } 
   else
   {
      D.f[DIR_M00] = &DD[DIR_P00 * numberOfLBnodes];
      D.f[DIR_P00] = &DD[DIR_M00 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real VeloX = vx[k];
      //real VeloY = vy[k];
      //real VeloZ = vz[k]; //(16.0*(u0*2.0)*bbx*bby*(grid_nx-bbx)*(grid_ny-bby))/(grid_nx*grid_nx*grid_ny*grid_ny)
      ////////////////////////////////////////////////////////////////////////////////
      //real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
      //      *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
      //      *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
      //      *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
      //      *q_dirBSE, *q_dirBNW; 
      //q_dirE   = &QQ[DIR_P00 * numberOfBCnodes];
      //q_dirW   = &QQ[DIR_M00 * numberOfBCnodes];
      //q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      //q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      //q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      //q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      //q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      //q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      //q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      //q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      //q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      //q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      //q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      //q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      //q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      //q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      //q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      //q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      //q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      //q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      //q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      //q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      //q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      //q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      //q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      //q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[DIR_P00])[ke   ];
      f_E    = (D.f[DIR_M00])[kw   ];
      f_S    = (D.f[DIR_0P0])[kn   ];
      f_N    = (D.f[DIR_0M0])[ks   ];
      f_B    = (D.f[DIR_00P])[kt   ];
      f_T    = (D.f[DIR_00M])[kb   ];
      f_SW   = (D.f[DIR_PP0])[kne  ];
      f_NE   = (D.f[DIR_MM0])[ksw  ];
      f_NW   = (D.f[DIR_PM0])[kse  ];
      f_SE   = (D.f[DIR_MP0])[knw  ];
      f_BW   = (D.f[DIR_P0P])[kte  ];
      f_TE   = (D.f[DIR_M0M])[kbw  ];
      f_TW   = (D.f[DIR_P0M])[kbe  ];
      f_BE   = (D.f[DIR_M0P])[ktw  ];
      f_BS   = (D.f[DIR_0PP])[ktn  ];
      f_TN   = (D.f[DIR_0MM])[kbs  ];
      f_TS   = (D.f[DIR_0PM])[kbn  ];
      f_BN   = (D.f[DIR_0MP])[kts  ];
      f_BSW  = (D.f[DIR_PPP])[ktne ];
      f_BNE  = (D.f[DIR_MMP])[ktsw ];
      f_BNW  = (D.f[DIR_PMP])[ktse ];
      f_BSE  = (D.f[DIR_MPP])[ktnw ];
      f_TSW  = (D.f[DIR_PPM])[kbne ];
      f_TNE  = (D.f[DIR_MMM])[kbsw ];
      f_TNW  = (D.f[DIR_PMM])[kbse ];
      f_TSE  = (D.f[DIR_MPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      // real vx2, vx3, feq, q;
      real vx1, drho;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[DIR_000])[kzero]); 

       vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                 (f_E - f_W)) / (c1o1 + drho); 
         

    //   vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
    //              ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
    //              (f_N - f_S)) / (c1o1 + drho); 

    //   vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
    //              (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
    //              (f_T - f_B)) / (c1o1 + drho); 

      //real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (one + drho);

	  real nu = c1o3 * (c1o1 / om1 - c1o2);
	  real qw = c1o1;
	  real uTau = sqrt(nu * (vx1 - VeloX) / qw);

	  if (abs(uTau)/nu>11){
	  uTau = vx1 * 0.41 / (log10(9.8 * uTau * qw / nu));
	  

	  
	  vx[k] = vx1 - uTau * uTau * qw / nu;
	  vx[k] = (vx[k]> 0.05) ? 0.05 : ((vx[k]< -0.05) ? -0.05 : vx[k] );  
	  }
	  else{ vx[k]=c0o1; }
	  //vy[k] = 0.01;							//Test...muss wieder raus
	  //vz[k] = 0.01;							//Test...muss wieder raus

   //   //////////////////////////////////////////////////////////////////////////
   //   if (isEvenTimestep==false)
   //   {
   //      D.f[DIR_P00] = &DD[DIR_P00 * size_Mat];
   //      D.f[DIR_M00] = &DD[DIR_M00 * size_Mat];
   //      D.f[DIR_0P0] = &DD[DIR_0P0 * size_Mat];
   //      D.f[DIR_0M0] = &DD[DIR_0M0 * size_Mat];
   //      D.f[DIR_00P] = &DD[DIR_00P * size_Mat];
   //      D.f[DIR_00M] = &DD[DIR_00M * size_Mat];
   //      D.f[DIR_PP0] = &DD[DIR_PP0 * size_Mat];
   //      D.f[DIR_MM0] = &DD[DIR_MM0 * size_Mat];
   //      D.f[DIR_PM0] = &DD[DIR_PM0 * size_Mat];
   //      D.f[DIR_MP0] = &DD[DIR_MP0 * size_Mat];
   //      D.f[DIR_P0P] = &DD[DIR_P0P * size_Mat];
   //      D.f[DIR_M0M] = &DD[DIR_M0M * size_Mat];
   //      D.f[DIR_P0M] = &DD[DIR_P0M * size_Mat];
   //      D.f[DIR_M0P] = &DD[DIR_M0P * size_Mat];
   //      D.f[DIR_0PP] = &DD[DIR_0PP * size_Mat];
   //      D.f[DIR_0MM] = &DD[DIR_0MM * size_Mat];
   //      D.f[DIR_0PM] = &DD[DIR_0PM * size_Mat];
   //      D.f[DIR_0MP] = &DD[DIR_0MP * size_Mat];
   //      D.f[DIR_000] = &DD[DIR_000 * size_Mat];
   //      D.f[DIR_PPP] = &DD[DIR_PPP * size_Mat];
   //      D.f[DIR_MMP] = &DD[DIR_MMP * size_Mat];
   //      D.f[DIR_PMP] = &DD[DIR_PMP * size_Mat];
   //      D.f[DIR_MPP] = &DD[DIR_MPP * size_Mat];
   //      D.f[DIR_PPM] = &DD[DIR_PPM * size_Mat];
   //      D.f[DIR_MMM] = &DD[DIR_MMM * size_Mat];
   //      D.f[DIR_PMM] = &DD[DIR_PMM * size_Mat];
   //      D.f[DIR_MPM] = &DD[DIR_MPM * size_Mat];
   //   } 
   //   else
   //   {
   //      D.f[DIR_M00] = &DD[DIR_P00 * size_Mat];
   //      D.f[DIR_P00] = &DD[DIR_M00 * size_Mat];
   //      D.f[DIR_0M0] = &DD[DIR_0P0 * size_Mat];
   //      D.f[DIR_0P0] = &DD[DIR_0M0 * size_Mat];
   //      D.f[DIR_00M] = &DD[DIR_00P * size_Mat];
   //      D.f[DIR_00P] = &DD[DIR_00M * size_Mat];
   //      D.f[DIR_MM0] = &DD[DIR_PP0 * size_Mat];
   //      D.f[DIR_PP0] = &DD[DIR_MM0 * size_Mat];
   //      D.f[DIR_MP0] = &DD[DIR_PM0 * size_Mat];
   //      D.f[DIR_PM0] = &DD[DIR_MP0 * size_Mat];
   //      D.f[DIR_M0M] = &DD[DIR_P0P * size_Mat];
   //      D.f[DIR_P0P] = &DD[DIR_M0M * size_Mat];
   //      D.f[DIR_M0P] = &DD[DIR_P0M * size_Mat];
   //      D.f[DIR_P0M] = &DD[DIR_M0P * size_Mat];
   //      D.f[DIR_0MM] = &DD[DIR_0PP * size_Mat];
   //      D.f[DIR_0PP] = &DD[DIR_0MM * size_Mat];
   //      D.f[DIR_0MP] = &DD[DIR_0PM * size_Mat];
   //      D.f[DIR_0PM] = &DD[DIR_0MP * size_Mat];
   //      D.f[DIR_000] = &DD[DIR_000 * size_Mat];
   //      D.f[DIR_PPP] = &DD[DIR_MMM * size_Mat];
   //      D.f[DIR_MMP] = &DD[DIR_PPM * size_Mat];
   //      D.f[DIR_PMP] = &DD[DIR_MPM * size_Mat];
   //      D.f[DIR_MPP] = &DD[DIR_PMM * size_Mat];
   //      D.f[DIR_PPM] = &DD[DIR_MMP * size_Mat];
   //      D.f[DIR_MMM] = &DD[DIR_PPP * size_Mat];
   //      D.f[DIR_PMM] = &DD[DIR_MPP * size_Mat];
   //      D.f[DIR_MPM] = &DD[DIR_PMP * size_Mat];
   //   }
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
   //   //Test
   //   //(D.f[DIR_000])[k]=c1o10;
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	  ////ToDo anders Klammern

   //   q = q_dirE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*( vx1        )*/+c9over2*( vx1        )*( vx1        ) * (one + drho)-cu_sq); 
   //      (D.f[DIR_M00])[kw]=(one-q)/(one+q)*(f_E-f_W+(f_E+f_W-two*feq*om1)/(one-om1))*c1o2+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[DIR_M00])[kw]=zero;
   //   }

   //   q = q_dirW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(-vx1        )*/+c9over2*(-vx1        )*(-vx1        ) * (one + drho)-cu_sq); 
   //      (D.f[DIR_P00])[ke]=(one-q)/(one+q)*(f_W-f_E+(f_W+f_E-two*feq*om1)/(one-om1))*c1o2+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[DIR_P00])[ke]=zero;
   //   }

   //   q = q_dirN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(    vx2     )*/+c9over2*(     vx2    )*(     vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[DIR_0M0])[ks]=(one-q)/(one+q)*(f_N-f_S+(f_N+f_S-two*feq*om1)/(one-om1))*c1o2+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[DIR_0M0])[ks]=zero;
   //   }

   //   q = q_dirS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(   -vx2     )*/+c9over2*(    -vx2    )*(    -vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[DIR_0P0])[kn]=(one-q)/(one+q)*(f_S-f_N+(f_S+f_N-two*feq*om1)/(one-om1))*c1o2+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[DIR_0P0])[kn]=zero;
   //   }

   //   q = q_dirT[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(         vx3)*/+c9over2*(         vx3)*(         vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_00M])[kb]=(one-q)/(one+q)*(f_T-f_B+(f_T+f_B-two*feq*om1)/(one-om1))*c1o2+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[DIR_00M])[kb]=one;
   //   }

   //   q = q_dirB[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(        -vx3)*/+c9over2*(        -vx3)*(        -vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_00P])[kt]=(one-q)/(one+q)*(f_B-f_T+(f_B+f_T-two*feq*om1)/(one-om1))*c1o2+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[DIR_00P])[kt]=zero;
   //   }

   //   q = q_dirNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*( vx1+vx2    )*/+c9over2*( vx1+vx2    )*( vx1+vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[DIR_MM0])[ksw]=(one-q)/(one+q)*(f_NE-f_SW+(f_NE+f_SW-two*feq*om1)/(one-om1))*c1o2+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_MM0])[ksw]=zero;
   //   }

   //   q = q_dirSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(-vx1-vx2    )*/+c9over2*(-vx1-vx2    )*(-vx1-vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[DIR_PP0])[kne]=(one-q)/(one+q)*(f_SW-f_NE+(f_SW+f_NE-two*feq*om1)/(one-om1))*c1o2+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_PP0])[kne]=zero;
   //   }

   //   q = q_dirSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*( vx1-vx2    )*/+c9over2*( vx1-vx2    )*( vx1-vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[DIR_MP0])[knw]=(one-q)/(one+q)*(f_SE-f_NW+(f_SE+f_NW-two*feq*om1)/(one-om1))*c1o2+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_MP0])[knw]=zero;
   //   }

   //   q = q_dirNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(-vx1+vx2    )*/+c9over2*(-vx1+vx2    )*(-vx1+vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[DIR_PM0])[kse]=(one-q)/(one+q)*(f_NW-f_SE+(f_NW+f_SE-two*feq*om1)/(one-om1))*c1o2+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_PM0])[kse]=zero;
   //   }

   //   q = q_dirTE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*( vx1    +vx3)*/+c9over2*( vx1    +vx3)*( vx1    +vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_M0M])[kbw]=(one-q)/(one+q)*(f_TE-f_BW+(f_TE+f_BW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_M0M])[kbw]=zero;
   //   }

   //   q = q_dirBW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(-vx1    -vx3)*/+c9over2*(-vx1    -vx3)*(-vx1    -vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_P0P])[kte]=(one-q)/(one+q)*(f_BW-f_TE+(f_BW+f_TE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_P0P])[kte]=zero;
   //   }

   //   q = q_dirBE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*( vx1    -vx3)*/+c9over2*( vx1    -vx3)*( vx1    -vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_M0P])[ktw]=(one-q)/(one+q)*(f_BE-f_TW+(f_BE+f_TW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_M0P])[ktw]=zero;
   //   }

   //   q = q_dirTW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(-vx1    +vx3)*/+c9over2*(-vx1    +vx3)*(-vx1    +vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_P0M])[kbe]=(one-q)/(one+q)*(f_TW-f_BE+(f_TW+f_BE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_P0M])[kbe]=zero;
   //   }

   //   q = q_dirTN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(     vx2+vx3)*/+c9over2*(     vx2+vx3)*(     vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_0MM])[kbs]=(one-q)/(one+q)*(f_TN-f_BS+(f_TN+f_BS-two*feq*om1)/(one-om1))*c1o2+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_0MM])[kbs]=zero;
   //   }

   //   q = q_dirBS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(    -vx2-vx3)*/+c9over2*(    -vx2-vx3)*(    -vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_0PP])[ktn]=(one-q)/(one+q)*(f_BS-f_TN+(f_BS+f_TN-two*feq*om1)/(one-om1))*c1o2+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_0PP])[ktn]=zero;
   //   }

   //   q = q_dirBN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(     vx2-vx3)*/+c9over2*(     vx2-vx3)*(     vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_0MP])[kts]=(one-q)/(one+q)*(f_BN-f_TS+(f_BN+f_TS-two*feq*om1)/(one-om1))*c1o2+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_0MP])[kts]=zero;
   //   }

   //   q = q_dirTS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(    -vx2+vx3)*/+c9over2*(    -vx2+vx3)*(    -vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_0PM])[kbn]=(one-q)/(one+q)*(f_TS-f_BN+(f_TS+f_BN-two*feq*om1)/(one-om1))*c1o2+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[DIR_0PM])[kbn]=zero;
   //   }

   //   q = q_dirTNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*( vx1+vx2+vx3)*/+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_MMM])[kbsw]=(one-q)/(one+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[DIR_MMM])[kbsw]=zero;
   //   }

   //   q = q_dirBSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*(-vx1-vx2-vx3)*/+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_PPP])[ktne]=(one-q)/(one+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[DIR_PPP])[ktne]=zero;
   //   }

   //   q = q_dirBNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*( vx1+vx2-vx3)*/+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_MMP])[ktsw]=(one-q)/(one+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[DIR_MMP])[ktsw]=zero;
   //   }

   //   q = q_dirTSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*(-vx1-vx2+vx3)*/+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_PPM])[kbne]=(one-q)/(one+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[DIR_PPM])[kbne]=zero;
   //   }

   //   q = q_dirTSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*( vx1-vx2+vx3)*/+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_MPM])[kbnw]=(one-q)/(one+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[DIR_MPM])[kbnw]=zero;
   //   }

   //   q = q_dirBNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*(-vx1+vx2-vx3)*/+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_PMP])[ktse]=(one-q)/(one+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[DIR_PMP])[ktse]=zero;
   //   }

   //   q = q_dirBSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*( vx1-vx2-vx3)*/+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_MPP])[ktnw]=(one-q)/(one+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[DIR_MPP])[ktnw]=zero;
   //   }

   //   q = q_dirTNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*(-vx1+vx2+vx3)*/+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[DIR_PMM])[kbse]=(one-q)/(one+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[DIR_PMM])[kbse]=zero;
   //   }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////









