#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;

//////////////////////////////////////////////////////////////////////////////
__global__ void InitParticles( real* coordX,
										  real* coordY,
										  real* coordZ, 
										  real* coordParticleXlocal,
										  real* coordParticleYlocal,
										  real* coordParticleZlocal,
										  real* coordParticleXglobal,
										  real* coordParticleYglobal,
										  real* coordParticleZglobal,
										  real* veloParticleX,
										  real* veloParticleY,
										  real* veloParticleZ,
										  real* randArray,
										  unsigned int* particleID,
										  unsigned int* cellBaseID,
										  unsigned int* bcMatD,
										  unsigned int* neighborX,
										  unsigned int* neighborY,
										  unsigned int* neighborZ,
										  unsigned int* neighborWSB,
										  int level,
									      unsigned int numberOfParticles, 
										  unsigned long long numberOfLBnodes)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k < numberOfParticles)
   {
	 //   real centerX = one;						//uebergabeparameter
	 //   real centerY = 10.5f;					//uebergabeparameter
		//real centerZ = 10.5f;					//uebergabeparameter
		//real diameter = 21.0f;					//uebergabeparameter
		//unsigned int numberOfParticleSizes = 41;	//uebergabeparameter
		//unsigned int nops = (unsigned int)(randArray[k]*numberOfParticleSizes);
		//real xCoordPart = one;
		//real yCoordPart = (real)(randArray[k]*diameter);
		//real zCoordPart = one;
		//if (k==0)
		//{
		//	zCoordPart = (real)(randArray[k+1]*diameter);
		//}
		//else
		//{
		//	zCoordPart = (real)(randArray[k-1]*diameter);
		//}
		//real distance = powf((zCoordPart-centerZ),2) + powf((yCoordPart-centerY),2);
		//real refDistance = powf((diameter*c1o2),2);
		//if (distance > refDistance)
		//{
		//	zCoordPart = sqrtf(powf((diameter*c1o2),2) - powf((yCoordPart-centerY),2)) + centerZ;
		//}



		////////////////////////////////////////////////////////////////////////////////
		//find random node of the fluid domain
		unsigned int cbID = (unsigned int)(randArray[k]*numberOfLBnodes);
		for(int i = 0; i < numberOfLBnodes;i++)
		{
			//if (coordX[cbID] < 15 && coordX[cbID] > 5 && coordY[cbID] < 15 && coordY[cbID] > 5 && coordZ[cbID] < 15 && coordZ[cbID] > 5)	break;
			if (coordX[cbID] < 5 && coordX[cbID] > 2)	break;
			cbID = (unsigned int)(randArray[k]*(numberOfLBnodes - i)); 
		}
	   
		real coordinateX;
		real coordinateY;
		real coordinateZ;

		unsigned int BC  = bcMatD[cbID];
		unsigned int BCx = bcMatD[neighborX[cbID]];
		unsigned int BCy = bcMatD[neighborY[cbID]];
		unsigned int BCz = bcMatD[neighborZ[cbID]];

		if( (BC == GEO_FLUID) && (BCx == GEO_FLUID) && (BCy == GEO_FLUID) && (BCz == GEO_FLUID))
		{
		   coordinateX = coordX[cbID];
		   coordinateY = coordY[cbID];
		   coordinateZ = coordZ[cbID];

		}
		else if(BC == GEO_FLUID)
		{
		   cbID = neighborWSB[neighborWSB[cbID]];
		   coordinateX = coordX[cbID];
		   coordinateY = coordY[cbID];
		   coordinateZ = coordZ[cbID];
		}
		else
		{
		   cbID = neighborZ[neighborY[neighborX[cbID]]];
		   coordinateX = coordX[cbID];
		   coordinateY = coordY[cbID];
		   coordinateZ = coordZ[cbID];
		}


		real localX = randArray[k] / (real)(pow((double)c2o1, (double)level));
        real localY = randArray[k] / (real)(pow((double)c2o1, (double)level));
        real localZ = randArray[k] / (real)(pow((double)c2o1, (double)level));

		real globalX = coordinateX + localX;
		real globalY = coordinateY + localY;
		real globalZ = coordinateZ + localZ;

  		real veloX = c0o1;
		real veloY = c0o1;
		real veloZ = c0o1;

		particleID[k]           = k      ;
		cellBaseID[k]           = cbID   ;
		veloParticleX[k]        = veloX  ;
		veloParticleY[k]        = veloY  ;
		veloParticleZ[k]        = veloZ  ;
		coordParticleXlocal[k]  = localX ;
		coordParticleYlocal[k]  = localY ;
		coordParticleZlocal[k]  = localZ ;
		coordParticleXglobal[k] = globalX;
		coordParticleYglobal[k] = globalY;
		coordParticleZglobal[k] = globalZ;
		////////////////////////////////////////////////////////////////////////////////
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


















//////////////////////////////////////////////////////////////////////////////
__global__ void MoveParticles( real* coordX,
										  real* coordY,
										  real* coordZ, 
										  real* coordParticleXlocal,
										  real* coordParticleYlocal,
										  real* coordParticleZlocal,
										  real* coordParticleXglobal,
										  real* coordParticleYglobal,
										  real* coordParticleZglobal,
										  real* veloParticleX,
										  real* veloParticleY,
										  real* veloParticleZ,
										  real* DD,
										  real  omega,
										  unsigned int* particleID,
										  unsigned int* cellBaseID,
										  unsigned int* bcMatD,
										  unsigned int* neighborX,
										  unsigned int* neighborY,
										  unsigned int* neighborZ,
										  unsigned int* neighborWSB,
										  int level,
										  unsigned int timestep, 
										  unsigned int numberOfTimesteps, 
									      unsigned int numberOfParticles, 
										  unsigned long long numberOfLBnodes,
										  bool isEvenTimestep)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  ix = threadIdx.x;  // Globaler x-Index 
   const unsigned  iy = blockIdx.x;   // Globaler y-Index 
   const unsigned  iz = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*iz + iy) + ix;
   //////////////////////////////////////////////////////////////////////////

    //real press;
   real vx1,vx2,vx3;
   real drho_SWT,vx1_SWT,vx2_SWT,vx3_SWT;
   real drho_NWT,vx1_NWT,vx2_NWT,vx3_NWT;
   real drho_NET,vx1_NET,vx2_NET,vx3_NET;
   real drho_SET,vx1_SET,vx2_SET,vx3_SET;
   real drho_SWB,vx1_SWB,vx2_SWB,vx3_SWB;
   real drho_NWB,vx1_NWB,vx2_NWB,vx3_NWB;
   real drho_NEB,vx1_NEB,vx2_NEB,vx3_NEB;
   real drho_SEB,vx1_SEB,vx2_SEB,vx3_SEB;
   real f_E,f_W,f_N,f_S,f_T,f_B,f_NE,f_SW,f_SE,f_NW,f_TE,f_BW,f_BE,f_TW,f_TN,f_BS,f_BN,f_TS,f_ZERO,f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;
   real *feC, *fwC, *fnC, *fsC, *ftC, *fbC, *fneC, *fswC, *fseC, *fnwC, *fteC, *fbwC, *fbeC, *ftwC, *ftnC, *fbsC, *fbnC, *ftsC, *fzeroC, *ftneC, *ftswC, *ftseC, *ftnwC, *fbneC, *fbswC, *fbseC, *fbnwC;
   real kxyFromfcNEQ_SWT, kyzFromfcNEQ_SWT, kxzFromfcNEQ_SWT, kxxMyyFromfcNEQ_SWT, kxxMzzFromfcNEQ_SWT;
   real kxyFromfcNEQ_NWT, kyzFromfcNEQ_NWT, kxzFromfcNEQ_NWT, kxxMyyFromfcNEQ_NWT, kxxMzzFromfcNEQ_NWT;
   real kxyFromfcNEQ_NET, kyzFromfcNEQ_NET, kxzFromfcNEQ_NET, kxxMyyFromfcNEQ_NET, kxxMzzFromfcNEQ_NET;
   real kxyFromfcNEQ_SET, kyzFromfcNEQ_SET, kxzFromfcNEQ_SET, kxxMyyFromfcNEQ_SET, kxxMzzFromfcNEQ_SET;
   real kxyFromfcNEQ_SWB, kyzFromfcNEQ_SWB, kxzFromfcNEQ_SWB, kxxMyyFromfcNEQ_SWB, kxxMzzFromfcNEQ_SWB;
   real kxyFromfcNEQ_NWB, kyzFromfcNEQ_NWB, kxzFromfcNEQ_NWB, kxxMyyFromfcNEQ_NWB, kxxMzzFromfcNEQ_NWB;
   real kxyFromfcNEQ_NEB, kyzFromfcNEQ_NEB, kxzFromfcNEQ_NEB, kxxMyyFromfcNEQ_NEB, kxxMzzFromfcNEQ_NEB;
   real kxyFromfcNEQ_SEB, kyzFromfcNEQ_SEB, kxzFromfcNEQ_SEB, kxxMyyFromfcNEQ_SEB, kxxMzzFromfcNEQ_SEB;
   real a0, ax, ay, az, axx, ayy, azz, axy, axz, ayz, b0, bx, by, bz, bxx, byy, bzz, bxy, bxz, byz, c0, cx, cy, cz, cxx, cyy, czz, cxy, cxz, cyz, axyz, bxyz, cxyz;
   //real d0, dx, dy, dz, dxy, dxz, dyz, dxyz;

   real x,y,z;

   if(k < numberOfParticles)
   {
		/////////////////////////////////////////////////////////////
	    unsigned int kTimeStep = k + (timestep * numberOfParticles);
		/////////////////////////////////////////////////////////////
		unsigned int kCellBaseID = cellBaseID[k];
	    unsigned int BC000  = bcMatD[kCellBaseID];
	    unsigned int BCx00  = bcMatD[neighborX[kCellBaseID]];
	    unsigned int BC0y0  = bcMatD[neighborY[kCellBaseID]];
	    unsigned int BC00z  = bcMatD[neighborZ[kCellBaseID]];
	    unsigned int BCxy0  = bcMatD[neighborY[neighborX[kCellBaseID]]];
	    unsigned int BCx0z  = bcMatD[neighborZ[neighborX[kCellBaseID]]];
	    unsigned int BC0yz  = bcMatD[neighborZ[neighborY[kCellBaseID]]];
	    unsigned int BCxyz  = bcMatD[neighborZ[neighborY[neighborX[kCellBaseID]]]];
		/////////////////////////////////////////////////////////////
   		if( (BC000 >= GEO_FLUID) || 
			(BCx00 >= GEO_FLUID) || 
			(BC0y0 >= GEO_FLUID) || 
			(BC00z >= GEO_FLUID) || 
			(BCxy0 >= GEO_FLUID) || 
			(BCx0z >= GEO_FLUID) || 
			(BC0yz >= GEO_FLUID) || 
			(BCxyz >= GEO_FLUID) )
		{
		   if (isEvenTimestep==true)
		   {
			  feC    = &DD[dP00 * numberOfLBnodes];
			  fwC    = &DD[dM00 * numberOfLBnodes];
			  fnC    = &DD[d0P0 * numberOfLBnodes];
			  fsC    = &DD[d0M0 * numberOfLBnodes];
			  ftC    = &DD[d00P * numberOfLBnodes];
			  fbC    = &DD[d00M * numberOfLBnodes];
			  fneC   = &DD[dPP0 * numberOfLBnodes];
			  fswC   = &DD[dMM0 * numberOfLBnodes];
			  fseC   = &DD[dPM0 * numberOfLBnodes];
			  fnwC   = &DD[dMP0 * numberOfLBnodes];
			  fteC   = &DD[dP0P * numberOfLBnodes];
			  fbwC   = &DD[dM0M * numberOfLBnodes];
			  fbeC   = &DD[dP0M * numberOfLBnodes];
			  ftwC   = &DD[dM0P * numberOfLBnodes];
			  ftnC   = &DD[d0PP * numberOfLBnodes];
			  fbsC   = &DD[d0MM * numberOfLBnodes];
			  fbnC   = &DD[d0PM * numberOfLBnodes];
			  ftsC   = &DD[d0MP * numberOfLBnodes];
			  fzeroC = &DD[d000 * numberOfLBnodes];
			  ftneC  = &DD[dPPP * numberOfLBnodes];
			  ftswC  = &DD[dMMP * numberOfLBnodes];
			  ftseC  = &DD[dPMP * numberOfLBnodes];
			  ftnwC  = &DD[dMPP * numberOfLBnodes];
			  fbneC  = &DD[dPPM * numberOfLBnodes];
			  fbswC  = &DD[dMMM * numberOfLBnodes];
			  fbseC  = &DD[dPMM * numberOfLBnodes];
			  fbnwC  = &DD[dMPM * numberOfLBnodes];
		   } 			 
		   else			 
		   {			 
			  fwC    = &DD[dP00 * numberOfLBnodes];
			  feC    = &DD[dM00 * numberOfLBnodes];
			  fsC    = &DD[d0P0 * numberOfLBnodes];
			  fnC    = &DD[d0M0 * numberOfLBnodes];
			  fbC    = &DD[d00P * numberOfLBnodes];
			  ftC    = &DD[d00M * numberOfLBnodes];
			  fswC   = &DD[dPP0 * numberOfLBnodes];
			  fneC   = &DD[dMM0 * numberOfLBnodes];
			  fnwC   = &DD[dPM0 * numberOfLBnodes];
			  fseC   = &DD[dMP0 * numberOfLBnodes];
			  fbwC   = &DD[dP0P * numberOfLBnodes];
			  fteC   = &DD[dM0M * numberOfLBnodes];
			  ftwC   = &DD[dP0M * numberOfLBnodes];
			  fbeC   = &DD[dM0P * numberOfLBnodes];
			  fbsC   = &DD[d0PP * numberOfLBnodes];
			  ftnC   = &DD[d0MM * numberOfLBnodes];
			  ftsC   = &DD[d0PM * numberOfLBnodes];
			  fbnC   = &DD[d0MP * numberOfLBnodes];
			  fzeroC = &DD[d000 * numberOfLBnodes];
			  fbswC  = &DD[dPPP * numberOfLBnodes];
			  fbneC  = &DD[dMMP * numberOfLBnodes];
			  fbnwC  = &DD[dPMP * numberOfLBnodes];
			  fbseC  = &DD[dMPP * numberOfLBnodes];
			  ftswC  = &DD[dPPM * numberOfLBnodes];
			  ftneC  = &DD[dMMM * numberOfLBnodes];
			  ftnwC  = &DD[dPMM * numberOfLBnodes];
			  ftseC  = &DD[dMPM * numberOfLBnodes];
		   }

			  //////////////////////////////////////////////////////////////////////////
			  //SWB//
			  //////////////////////////////////////////////////////////////////////////
			  //index 0
			  unsigned int k0zero= cellBaseID[k];
			  unsigned int k0w   = neighborX[k0zero];
			  unsigned int k0s   = neighborY[k0zero];
			  unsigned int k0b   = neighborZ[k0zero];
			  unsigned int k0sw  = neighborY[k0w];
			  unsigned int k0bw  = neighborZ[k0w];
			  unsigned int k0bs  = neighborZ[k0s];
			  unsigned int k0bsw = neighborZ[k0sw];
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  unsigned int kzero= k0zero;
			  unsigned int kw   = k0w;   
			  unsigned int ks   = k0s;   
			  unsigned int kb   = k0b;   
			  unsigned int ksw  = k0sw;  
			  unsigned int kbw  = k0bw;  
			  unsigned int kbs  = k0bs;  
			  unsigned int kbsw = k0bsw; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_SWB = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_SWB  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_SWB);
			  vx2_SWB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_SWB);
			  vx3_SWB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_SWB);

			  kxyFromfcNEQ_SWB    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_SWB) - ((vx1_SWB*vx2_SWB)));
			  kyzFromfcNEQ_SWB    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_SWB) - ((vx2_SWB*vx3_SWB)));
			  kxzFromfcNEQ_SWB    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_SWB) - ((vx1_SWB*vx3_SWB)));
			  kxxMyyFromfcNEQ_SWB = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_SWB) - ((vx1_SWB*vx1_SWB-vx2_SWB*vx2_SWB)));
			  kxxMzzFromfcNEQ_SWB = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_SWB) - ((vx1_SWB*vx1_SWB-vx3_SWB*vx3_SWB)));

			  //////////////////////////////////////////////////////////////////////////
			  //SWT//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= kb;
			  kw   = kbw;   
			  ks   = kbs;   
			  kb   = neighborZ[kb];   
			  ksw  = kbsw;  
			  kbw  = neighborZ[kbw];  
			  kbs  = neighborZ[kbs];  
			  kbsw = neighborZ[kbsw]; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_SWT = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_SWT  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_SWT);
			  vx2_SWT  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_SWT);
			  vx3_SWT  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_SWT);

			  kxyFromfcNEQ_SWT    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_SWT) - ((vx1_SWT*vx2_SWT)));
			  kyzFromfcNEQ_SWT    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_SWT) - ((vx2_SWT*vx3_SWT)));
			  kxzFromfcNEQ_SWT    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_SWT) - ((vx1_SWT*vx3_SWT)));
			  kxxMyyFromfcNEQ_SWT = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_SWT) - ((vx1_SWT*vx1_SWT-vx2_SWT*vx2_SWT)));
			  kxxMzzFromfcNEQ_SWT = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_SWT) - ((vx1_SWT*vx1_SWT-vx3_SWT*vx3_SWT)));

			  //////////////////////////////////////////////////////////////////////////
			  //SET//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= kw;
			  kw   = neighborX[kw];   
			  ks   = ksw;   
			  kb   = kbw;   
			  ksw  = neighborX[ksw];  
			  kbw  = neighborX[kbw];  
			  kbs  = kbsw;  
			  kbsw = neighborX[kbsw]; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_SET = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_SET  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_SET);
			  vx2_SET  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_SET);
			  vx3_SET  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_SET);

			  kxyFromfcNEQ_SET    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_SET) - ((vx1_SET*vx2_SET)));
			  kyzFromfcNEQ_SET    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_SET) - ((vx2_SET*vx3_SET)));
			  kxzFromfcNEQ_SET    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_SET) - ((vx1_SET*vx3_SET)));
			  kxxMyyFromfcNEQ_SET = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_SET) - ((vx1_SET*vx1_SET-vx2_SET*vx2_SET)));
			  kxxMzzFromfcNEQ_SET = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_SET) - ((vx1_SET*vx1_SET-vx3_SET*vx3_SET)));

			  //////////////////////////////////////////////////////////////////////////
			  //SEB//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kb   = kzero;   
			  kbw  = kw;  
			  kbs  = ks;  
			  kbsw = ksw; 
			  kzero= k0w;
			  kw   = neighborX[k0w];   
			  ks   = k0sw;   
			  ksw  = neighborX[k0sw];  
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_SEB = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_SEB  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_SEB);
			  vx2_SEB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_SEB);
			  vx3_SEB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_SEB);

			  kxyFromfcNEQ_SEB    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_SEB) - ((vx1_SEB*vx2_SEB)));
			  kyzFromfcNEQ_SEB    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_SEB) - ((vx2_SEB*vx3_SEB)));
			  kxzFromfcNEQ_SEB    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_SEB) - ((vx1_SEB*vx3_SEB)));
			  kxxMyyFromfcNEQ_SEB = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_SEB) - ((vx1_SEB*vx1_SEB-vx2_SEB*vx2_SEB)));
			  kxxMzzFromfcNEQ_SEB = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_SEB) - ((vx1_SEB*vx1_SEB-vx3_SEB*vx3_SEB)));

			  //////////////////////////////////////////////////////////////////////////
			  //NWB//
			  //////////////////////////////////////////////////////////////////////////
			  //index 0
			  k0zero= k0s;
			  k0w   = k0sw;
			  k0s   = neighborY[k0s];
			  k0b   = k0bs;
			  k0sw  = neighborY[k0sw];
			  k0bw  = k0bsw;
			  k0bs  = neighborY[k0bs];
			  k0bsw = neighborY[k0bsw];
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= k0zero;
			  kw   = k0w;   
			  ks   = k0s;   
			  kb   = k0b;   
			  ksw  = k0sw;  
			  kbw  = k0bw;  
			  kbs  = k0bs;  
			  kbsw = k0bsw; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_NWB = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_NWB  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_NWB);
			  vx2_NWB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_NWB);
			  vx3_NWB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_NWB);

			  kxyFromfcNEQ_NWB    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_NWB) - ((vx1_NWB*vx2_NWB)));
			  kyzFromfcNEQ_NWB    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_NWB) - ((vx2_NWB*vx3_NWB)));
			  kxzFromfcNEQ_NWB    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_NWB) - ((vx1_NWB*vx3_NWB)));
			  kxxMyyFromfcNEQ_NWB = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_NWB) - ((vx1_NWB*vx1_NWB-vx2_NWB*vx2_NWB)));
			  kxxMzzFromfcNEQ_NWB = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_NWB) - ((vx1_NWB*vx1_NWB-vx3_NWB*vx3_NWB)));

			  //////////////////////////////////////////////////////////////////////////
			  //NWT//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= kb;
			  kw   = kbw;   
			  ks   = kbs;   
			  kb   = neighborZ[kb];   
			  ksw  = kbsw;  
			  kbw  = neighborZ[kbw];  
			  kbs  = neighborZ[kbs];  
			  kbsw = neighborZ[kbsw]; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_NWT = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_NWT  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_NWT);
			  vx2_NWT  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_NWT);
			  vx3_NWT  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_NWT);

			  kxyFromfcNEQ_NWT    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_NWT) - ((vx1_NWT*vx2_NWT)));
			  kyzFromfcNEQ_NWT    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_NWT) - ((vx2_NWT*vx3_NWT)));
			  kxzFromfcNEQ_NWT    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_NWT) - ((vx1_NWT*vx3_NWT)));
			  kxxMyyFromfcNEQ_NWT = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_NWT) - ((vx1_NWT*vx1_NWT-vx2_NWT*vx2_NWT)));
			  kxxMzzFromfcNEQ_NWT = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_NWT) - ((vx1_NWT*vx1_NWT-vx3_NWT*vx3_NWT)));

			  //////////////////////////////////////////////////////////////////////////
			  //NET//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= kw;
			  kw   = neighborX[kw];   
			  ks   = ksw;   
			  kb   = kbw;   
			  ksw  = neighborX[ksw];  
			  kbw  = neighborX[kbw];  
			  kbs  = kbsw;  
			  kbsw = neighborX[kbsw]; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_NET = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_NET  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_NET);
			  vx2_NET  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_NET);
			  vx3_NET  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_NET);

			  kxyFromfcNEQ_NET    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_NET) - ((vx1_NET*vx2_NET)));
			  kyzFromfcNEQ_NET    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_NET) - ((vx2_NET*vx3_NET)));
			  kxzFromfcNEQ_NET    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_NET) - ((vx1_NET*vx3_NET)));
			  kxxMyyFromfcNEQ_NET = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_NET) - ((vx1_NET*vx1_NET-vx2_NET*vx2_NET)));
			  kxxMzzFromfcNEQ_NET = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_NET) - ((vx1_NET*vx1_NET-vx3_NET*vx3_NET)));

			  //////////////////////////////////////////////////////////////////////////
			  //NEB//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kb   = kzero;   
			  kbw  = kw;  
			  kbs  = ks;  
			  kbsw = ksw; 
			  kzero= k0w;
			  kw   = neighborX[k0w];   
			  ks   = k0sw;   
			  ksw  = neighborX[k0sw];  
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_NEB = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_NEB  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_NEB);
			  vx2_NEB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_NEB);
			  vx3_NEB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_NEB);

			  kxyFromfcNEQ_NEB    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_NEB) - ((vx1_NEB*vx2_NEB)));
			  kyzFromfcNEQ_NEB    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_NEB) - ((vx2_NEB*vx3_NEB)));
			  kxzFromfcNEQ_NEB    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_NEB) - ((vx1_NEB*vx3_NEB)));
			  kxxMyyFromfcNEQ_NEB = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_NEB) - ((vx1_NEB*vx1_NEB-vx2_NEB*vx2_NEB)));
			  kxxMzzFromfcNEQ_NEB = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_NEB) - ((vx1_NEB*vx1_NEB-vx3_NEB*vx3_NEB)));

			  //////////////////////////////////////////////////////////////////////////
			  //interpolate
			  //////////////////////////////////////////////////////////////////////////
			  a0 = (-kxxMyyFromfcNEQ_NEB - kxxMyyFromfcNEQ_NET + kxxMyyFromfcNEQ_NWB + kxxMyyFromfcNEQ_NWT - 
				 kxxMyyFromfcNEQ_SEB - kxxMyyFromfcNEQ_SET + kxxMyyFromfcNEQ_SWB + kxxMyyFromfcNEQ_SWT - 
				 kxxMzzFromfcNEQ_NEB - kxxMzzFromfcNEQ_NET + kxxMzzFromfcNEQ_NWB + kxxMzzFromfcNEQ_NWT - 
				 kxxMzzFromfcNEQ_SEB - kxxMzzFromfcNEQ_SET + kxxMzzFromfcNEQ_SWB + kxxMzzFromfcNEQ_SWT - 
				 c2o1*kxyFromfcNEQ_NEB - c2o1*kxyFromfcNEQ_NET - c2o1*kxyFromfcNEQ_NWB - c2o1*kxyFromfcNEQ_NWT + 
				 c2o1*kxyFromfcNEQ_SEB + c2o1*kxyFromfcNEQ_SET + c2o1*kxyFromfcNEQ_SWB + c2o1*kxyFromfcNEQ_SWT + 
				 c2o1*kxzFromfcNEQ_NEB - c2o1*kxzFromfcNEQ_NET + c2o1*kxzFromfcNEQ_NWB - c2o1*kxzFromfcNEQ_NWT + 
				 c2o1*kxzFromfcNEQ_SEB - c2o1*kxzFromfcNEQ_SET + c2o1*kxzFromfcNEQ_SWB - c2o1*kxzFromfcNEQ_SWT + 
				 c8o1*vx1_NEB + c8o1*vx1_NET + c8o1*vx1_NWB + c8o1*vx1_NWT + c8o1*vx1_SEB + 
				 c8o1*vx1_SET + c8o1*vx1_SWB + c8o1*vx1_SWT + c2o1*vx2_NEB + c2o1*vx2_NET - 
				 c2o1*vx2_NWB - c2o1*vx2_NWT - c2o1*vx2_SEB - c2o1*vx2_SET + c2o1*vx2_SWB + 
				 c2o1*vx2_SWT - c2o1*vx3_NEB + c2o1*vx3_NET + c2o1*vx3_NWB - c2o1*vx3_NWT - 
				 c2o1*vx3_SEB + c2o1*vx3_SET + c2o1*vx3_SWB - c2o1*vx3_SWT)/c64o1;
			  b0 = (c2o1*kxxMyyFromfcNEQ_NEB + c2o1*kxxMyyFromfcNEQ_NET + c2o1*kxxMyyFromfcNEQ_NWB + c2o1*kxxMyyFromfcNEQ_NWT - 
				 c2o1*kxxMyyFromfcNEQ_SEB - c2o1*kxxMyyFromfcNEQ_SET - c2o1*kxxMyyFromfcNEQ_SWB - c2o1*kxxMyyFromfcNEQ_SWT - 
				 kxxMzzFromfcNEQ_NEB - kxxMzzFromfcNEQ_NET - kxxMzzFromfcNEQ_NWB - kxxMzzFromfcNEQ_NWT + 
				 kxxMzzFromfcNEQ_SEB + kxxMzzFromfcNEQ_SET + kxxMzzFromfcNEQ_SWB + kxxMzzFromfcNEQ_SWT - 
				 c2o1*kxyFromfcNEQ_NEB - c2o1*kxyFromfcNEQ_NET + c2o1*kxyFromfcNEQ_NWB + c2o1*kxyFromfcNEQ_NWT - 
				 c2o1*kxyFromfcNEQ_SEB - c2o1*kxyFromfcNEQ_SET + c2o1*kxyFromfcNEQ_SWB + c2o1*kxyFromfcNEQ_SWT + 
				 c2o1*kyzFromfcNEQ_NEB - c2o1*kyzFromfcNEQ_NET + c2o1*kyzFromfcNEQ_NWB - c2o1*kyzFromfcNEQ_NWT + 
				 c2o1*kyzFromfcNEQ_SEB - c2o1*kyzFromfcNEQ_SET + c2o1*kyzFromfcNEQ_SWB - c2o1*kyzFromfcNEQ_SWT + 
				 c2o1*vx1_NEB + c2o1*vx1_NET - c2o1*vx1_NWB - c2o1*vx1_NWT - 
				 c2o1*vx1_SEB - c2o1*vx1_SET + c2o1*vx1_SWB + c2o1*vx1_SWT + 
				 c8o1*vx2_NEB + c8o1*vx2_NET + c8o1*vx2_NWB + c8o1*vx2_NWT + 
				 c8o1*vx2_SEB + c8o1*vx2_SET + c8o1*vx2_SWB + c8o1*vx2_SWT - 
				 c2o1*vx3_NEB + c2o1*vx3_NET - c2o1*vx3_NWB + c2o1*vx3_NWT + 
				 c2o1*vx3_SEB - c2o1*vx3_SET + c2o1*vx3_SWB - c2o1*vx3_SWT)/c64o1;

			  //b0 = ((eight*vx2_NEB + eight*vx2_SWT) + (eight*vx2_NET + eight*vx2_SWB) + (eight*vx2_NWB + eight*vx2_SET) + (eight*vx2_NWT + eight*vx2_SEB))/sixtyfour;

			  c0 = (kxxMyyFromfcNEQ_NEB - kxxMyyFromfcNEQ_NET + kxxMyyFromfcNEQ_NWB - kxxMyyFromfcNEQ_NWT + 
				 kxxMyyFromfcNEQ_SEB - kxxMyyFromfcNEQ_SET + kxxMyyFromfcNEQ_SWB - kxxMyyFromfcNEQ_SWT - 
				 c2o1*kxxMzzFromfcNEQ_NEB + c2o1*kxxMzzFromfcNEQ_NET - c2o1*kxxMzzFromfcNEQ_NWB + c2o1*kxxMzzFromfcNEQ_NWT - 
				 c2o1*kxxMzzFromfcNEQ_SEB + c2o1*kxxMzzFromfcNEQ_SET - c2o1*kxxMzzFromfcNEQ_SWB + c2o1*kxxMzzFromfcNEQ_SWT - 
				 c2o1*kxzFromfcNEQ_NEB - c2o1*kxzFromfcNEQ_NET + c2o1*kxzFromfcNEQ_NWB + c2o1*kxzFromfcNEQ_NWT - 
				 c2o1*kxzFromfcNEQ_SEB - c2o1*kxzFromfcNEQ_SET + c2o1*kxzFromfcNEQ_SWB + c2o1*kxzFromfcNEQ_SWT - 
				 c2o1*kyzFromfcNEQ_NEB - c2o1*kyzFromfcNEQ_NET - c2o1*kyzFromfcNEQ_NWB - c2o1*kyzFromfcNEQ_NWT + 
				 c2o1*kyzFromfcNEQ_SEB + c2o1*kyzFromfcNEQ_SET + c2o1*kyzFromfcNEQ_SWB + c2o1*kyzFromfcNEQ_SWT - 
				 c2o1*vx1_NEB + c2o1*vx1_NET + c2o1*vx1_NWB - c2o1*vx1_NWT - 
				 c2o1*vx1_SEB + c2o1*vx1_SET + c2o1*vx1_SWB - c2o1*vx1_SWT - 
				 c2o1*vx2_NEB + c2o1*vx2_NET - c2o1*vx2_NWB + c2o1*vx2_NWT + 
				 c2o1*vx2_SEB - c2o1*vx2_SET + c2o1*vx2_SWB - c2o1*vx2_SWT + 
				 c8o1*vx3_NEB + c8o1*vx3_NET + c8o1*vx3_NWB + c8o1*vx3_NWT + 
				 c8o1*vx3_SEB + c8o1*vx3_SET + c8o1*vx3_SWB + c8o1*vx3_SWT)/c64o1;
			  ax = (vx1_NEB + vx1_NET - vx1_NWB - vx1_NWT + vx1_SEB + vx1_SET - vx1_SWB - vx1_SWT)/c4o1;
			  bx = (vx2_NEB + vx2_NET - vx2_NWB - vx2_NWT + vx2_SEB + vx2_SET - vx2_SWB - vx2_SWT)/c4o1;
			  //bx = ((vx2_NEB - vx2_SWT) + (vx2_NET - vx2_SWB) + (vx2_SET - vx2_NWB) + (vx2_SEB - vx2_NWT))/four;
			  cx = (vx3_NEB + vx3_NET - vx3_NWB - vx3_NWT + vx3_SEB + vx3_SET - vx3_SWB - vx3_SWT)/c4o1;
			  axx= (kxxMyyFromfcNEQ_NEB + kxxMyyFromfcNEQ_NET - kxxMyyFromfcNEQ_NWB - kxxMyyFromfcNEQ_NWT + 
				 kxxMyyFromfcNEQ_SEB + kxxMyyFromfcNEQ_SET - kxxMyyFromfcNEQ_SWB - kxxMyyFromfcNEQ_SWT + 
				 kxxMzzFromfcNEQ_NEB + kxxMzzFromfcNEQ_NET - kxxMzzFromfcNEQ_NWB - kxxMzzFromfcNEQ_NWT + 
				 kxxMzzFromfcNEQ_SEB + kxxMzzFromfcNEQ_SET - kxxMzzFromfcNEQ_SWB - kxxMzzFromfcNEQ_SWT + 
				 c2o1*vx2_NEB + c2o1*vx2_NET - c2o1*vx2_NWB - c2o1*vx2_NWT - 
				 c2o1*vx2_SEB - c2o1*vx2_SET + c2o1*vx2_SWB + c2o1*vx2_SWT - 
				 c2o1*vx3_NEB + c2o1*vx3_NET + c2o1*vx3_NWB - c2o1*vx3_NWT - 
				 c2o1*vx3_SEB + c2o1*vx3_SET + c2o1*vx3_SWB - c2o1*vx3_SWT)/c16o1;
			  bxx= (kxyFromfcNEQ_NEB + kxyFromfcNEQ_NET - kxyFromfcNEQ_NWB - kxyFromfcNEQ_NWT + 
				 kxyFromfcNEQ_SEB + kxyFromfcNEQ_SET - kxyFromfcNEQ_SWB - kxyFromfcNEQ_SWT - 
				 c2o1*vx1_NEB - c2o1*vx1_NET + c2o1*vx1_NWB + c2o1*vx1_NWT + 
				 c2o1*vx1_SEB + c2o1*vx1_SET - c2o1*vx1_SWB - c2o1*vx1_SWT)/c8o1;
			  cxx= (kxzFromfcNEQ_NEB + kxzFromfcNEQ_NET - kxzFromfcNEQ_NWB - kxzFromfcNEQ_NWT + 
				 kxzFromfcNEQ_SEB + kxzFromfcNEQ_SET - kxzFromfcNEQ_SWB - kxzFromfcNEQ_SWT + 
				 c2o1*vx1_NEB - c2o1*vx1_NET - c2o1*vx1_NWB + c2o1*vx1_NWT + 
				 c2o1*vx1_SEB - c2o1*vx1_SET - c2o1*vx1_SWB + c2o1*vx1_SWT)/c8o1;
			  ay = (vx1_NEB + vx1_NET + vx1_NWB + vx1_NWT - vx1_SEB - vx1_SET - vx1_SWB - vx1_SWT)/c4o1;
			  by = (vx2_NEB + vx2_NET + vx2_NWB + vx2_NWT - vx2_SEB - vx2_SET - vx2_SWB - vx2_SWT)/c4o1;
			  cy = (vx3_NEB + vx3_NET + vx3_NWB + vx3_NWT - vx3_SEB - vx3_SET - vx3_SWB - vx3_SWT)/c4o1;
			  ayy= (kxyFromfcNEQ_NEB + kxyFromfcNEQ_NET + kxyFromfcNEQ_NWB + kxyFromfcNEQ_NWT - 
				 kxyFromfcNEQ_SEB - kxyFromfcNEQ_SET - kxyFromfcNEQ_SWB - kxyFromfcNEQ_SWT - 
				 c2o1*vx2_NEB - c2o1*vx2_NET + c2o1*vx2_NWB + c2o1*vx2_NWT + 
				 c2o1*vx2_SEB + c2o1*vx2_SET - c2o1*vx2_SWB - c2o1*vx2_SWT)/c8o1;
			  byy= (-c2o1*kxxMyyFromfcNEQ_NEB - c2o1*kxxMyyFromfcNEQ_NET - c2o1*kxxMyyFromfcNEQ_NWB - c2o1*kxxMyyFromfcNEQ_NWT + 
				 c2o1*kxxMyyFromfcNEQ_SEB + c2o1*kxxMyyFromfcNEQ_SET + c2o1*kxxMyyFromfcNEQ_SWB + c2o1*kxxMyyFromfcNEQ_SWT + 
				 kxxMzzFromfcNEQ_NEB + kxxMzzFromfcNEQ_NET + kxxMzzFromfcNEQ_NWB + kxxMzzFromfcNEQ_NWT - 
				 kxxMzzFromfcNEQ_SEB - kxxMzzFromfcNEQ_SET - kxxMzzFromfcNEQ_SWB - kxxMzzFromfcNEQ_SWT + 
				 c2o1*vx1_NEB + c2o1*vx1_NET - c2o1*vx1_NWB - c2o1*vx1_NWT - 
				 c2o1*vx1_SEB - c2o1*vx1_SET + c2o1*vx1_SWB + c2o1*vx1_SWT - 
				 c2o1*vx3_NEB + c2o1*vx3_NET - c2o1*vx3_NWB + c2o1*vx3_NWT + 
				 c2o1*vx3_SEB - c2o1*vx3_SET + c2o1*vx3_SWB - c2o1*vx3_SWT)/c16o1;
			  cyy= (kyzFromfcNEQ_NEB + kyzFromfcNEQ_NET + kyzFromfcNEQ_NWB + kyzFromfcNEQ_NWT - 
				 kyzFromfcNEQ_SEB - kyzFromfcNEQ_SET - kyzFromfcNEQ_SWB - kyzFromfcNEQ_SWT + 
				 c2o1*vx2_NEB - c2o1*vx2_NET + c2o1*vx2_NWB - c2o1*vx2_NWT - 
				 c2o1*vx2_SEB + c2o1*vx2_SET - c2o1*vx2_SWB + c2o1*vx2_SWT)/c8o1;
			  az = (-vx1_NEB + vx1_NET - vx1_NWB + vx1_NWT - vx1_SEB + vx1_SET - vx1_SWB + vx1_SWT)/c4o1;
			  //bz = (-vx2_NEB + vx2_NET - vx2_NWB + vx2_NWT - vx2_SEB + vx2_SET - vx2_SWB + vx2_SWT)/four;
			  bz = ((vx2_SWT - vx2_NEB) + (vx2_NET - vx2_SWB) + (vx2_SET - vx2_NWB) + (vx2_NWT - vx2_SEB))/c4o1;
			  cz = (-vx3_NEB + vx3_NET - vx3_NWB + vx3_NWT - vx3_SEB + vx3_SET - vx3_SWB + vx3_SWT)/c4o1;
			  azz= (-kxzFromfcNEQ_NEB + kxzFromfcNEQ_NET - kxzFromfcNEQ_NWB + kxzFromfcNEQ_NWT - 
				 kxzFromfcNEQ_SEB + kxzFromfcNEQ_SET - kxzFromfcNEQ_SWB + kxzFromfcNEQ_SWT + 
				 c2o1*vx3_NEB - c2o1*vx3_NET - c2o1*vx3_NWB + c2o1*vx3_NWT + 
				 c2o1*vx3_SEB - c2o1*vx3_SET - c2o1*vx3_SWB + c2o1*vx3_SWT)/c8o1;
			  bzz= (-kyzFromfcNEQ_NEB + kyzFromfcNEQ_NET - kyzFromfcNEQ_NWB + kyzFromfcNEQ_NWT - 
				 kyzFromfcNEQ_SEB + kyzFromfcNEQ_SET - kyzFromfcNEQ_SWB + kyzFromfcNEQ_SWT + 
				 c2o1*vx3_NEB - c2o1*vx3_NET + c2o1*vx3_NWB - c2o1*vx3_NWT - 
				 c2o1*vx3_SEB + c2o1*vx3_SET - c2o1*vx3_SWB + c2o1*vx3_SWT)/c8o1;
			  czz= (-kxxMyyFromfcNEQ_NEB + kxxMyyFromfcNEQ_NET - kxxMyyFromfcNEQ_NWB + kxxMyyFromfcNEQ_NWT - 
				 kxxMyyFromfcNEQ_SEB + kxxMyyFromfcNEQ_SET - kxxMyyFromfcNEQ_SWB + kxxMyyFromfcNEQ_SWT + 
				 c2o1*kxxMzzFromfcNEQ_NEB - c2o1*kxxMzzFromfcNEQ_NET + c2o1*kxxMzzFromfcNEQ_NWB - c2o1*kxxMzzFromfcNEQ_NWT + 
				 c2o1*kxxMzzFromfcNEQ_SEB - c2o1*kxxMzzFromfcNEQ_SET + c2o1*kxxMzzFromfcNEQ_SWB - c2o1*kxxMzzFromfcNEQ_SWT - 
				 c2o1*vx1_NEB + c2o1*vx1_NET + c2o1*vx1_NWB - c2o1*vx1_NWT - 
				 c2o1*vx1_SEB + c2o1*vx1_SET + c2o1*vx1_SWB - c2o1*vx1_SWT - 
				 c2o1*vx2_NEB + c2o1*vx2_NET - c2o1*vx2_NWB + c2o1*vx2_NWT + 
				 c2o1*vx2_SEB - c2o1*vx2_SET + c2o1*vx2_SWB - c2o1*vx2_SWT)/c16o1;
			  axy= (vx1_NEB + vx1_NET - vx1_NWB - vx1_NWT - vx1_SEB - vx1_SET + vx1_SWB + vx1_SWT)/c2o1;
			  bxy= (vx2_NEB + vx2_NET - vx2_NWB - vx2_NWT - vx2_SEB - vx2_SET + vx2_SWB + vx2_SWT)/c2o1;
			  cxy= (vx3_NEB + vx3_NET - vx3_NWB - vx3_NWT - vx3_SEB - vx3_SET + vx3_SWB + vx3_SWT)/c2o1;
			  axz= (-vx1_NEB + vx1_NET + vx1_NWB - vx1_NWT - vx1_SEB + vx1_SET + vx1_SWB - vx1_SWT)/c2o1;
			  bxz= (-vx2_NEB + vx2_NET + vx2_NWB - vx2_NWT - vx2_SEB + vx2_SET + vx2_SWB - vx2_SWT)/c2o1;
			  cxz= (-vx3_NEB + vx3_NET + vx3_NWB - vx3_NWT - vx3_SEB + vx3_SET + vx3_SWB - vx3_SWT)/c2o1;
			  ayz= (-vx1_NEB + vx1_NET - vx1_NWB + vx1_NWT + vx1_SEB - vx1_SET + vx1_SWB - vx1_SWT)/c2o1;
			  byz= (-vx2_NEB + vx2_NET - vx2_NWB + vx2_NWT + vx2_SEB - vx2_SET + vx2_SWB - vx2_SWT)/c2o1;
			  cyz= (-vx3_NEB + vx3_NET - vx3_NWB + vx3_NWT + vx3_SEB - vx3_SET + vx3_SWB - vx3_SWT)/c2o1;
			  axyz=-vx1_NEB + vx1_NET + vx1_NWB - vx1_NWT + vx1_SEB - vx1_SET - vx1_SWB + vx1_SWT;
			  bxyz=-vx2_NEB + vx2_NET + vx2_NWB - vx2_NWT + vx2_SEB - vx2_SET - vx2_SWB + vx2_SWT;
			  cxyz=-vx3_NEB + vx3_NET + vx3_NWB - vx3_NWT + vx3_SEB - vx3_SET - vx3_SWB + vx3_SWT;
			  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  //drho
			//   d0   = ( drho_NEB + drho_NET + drho_NWB + drho_NWT + drho_SEB + drho_SET + drho_SWB + drho_SWT) * c1o8;
			//   dx   = ( drho_NEB + drho_NET - drho_NWB - drho_NWT + drho_SEB + drho_SET - drho_SWB - drho_SWT) * c1o4;
			//   dy   = ( drho_NEB + drho_NET + drho_NWB + drho_NWT - drho_SEB - drho_SET - drho_SWB - drho_SWT) * c1o4;
			//   dz   = (-drho_NEB + drho_NET - drho_NWB + drho_NWT - drho_SEB + drho_SET - drho_SWB + drho_SWT) * c1o4;
			//   dxy  = ( drho_NEB + drho_NET - drho_NWB - drho_NWT - drho_SEB - drho_SET + drho_SWB + drho_SWT) * c1o2;
			//   dxz  = (-drho_NEB + drho_NET + drho_NWB - drho_NWT - drho_SEB + drho_SET + drho_SWB - drho_SWT) * c1o2;
			//   dyz  = (-drho_NEB + drho_NET - drho_NWB + drho_NWT + drho_SEB - drho_SET + drho_SWB - drho_SWT) * c1o2;
			//   dxyz =  -drho_NEB + drho_NET + drho_NWB - drho_NWT + drho_SEB - drho_SET - drho_SWB + drho_SWT;
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  unsigned int kTimeStepOld = kTimeStep - numberOfParticles;
			  real localX = coordParticleXlocal[kTimeStepOld];
			  real localY = coordParticleYlocal[kTimeStepOld];
			  real localZ = coordParticleZlocal[kTimeStepOld];

			  x = (localX * (real)(pow((double)c2o1, (double)level))) - c1o2; //-c1o4;
              y = (localY * (real)(pow((double)c2o1, (double)level))) - c1o2; //-c1o4;
              z = (localZ * (real)(pow((double)c2o1, (double)level))) - c1o2; //-c1o4;
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  //press = d0 + x*dx + y*dy + z*dz + x*y*dxy + x*z*dxz + y*z*dyz + x*y*z*dxyz;
			  vx1 = (a0 + x*ax + y*ay + z*az + x*x*axx + y*y*ayy + z*z*azz + x*y*axy + x*z*axz + y*z*ayz + x*y*z*axyz);
			  vx2 = (b0 + x*bx + y*by + z*bz + x*x*bxx + y*y*byy + z*z*bzz + x*y*bxy + x*z*bxz + y*z*byz + x*y*z*bxyz);
			  vx3 = (c0 + x*cx + y*cy + z*cz + x*x*cxx + y*y*cyy + z*z*czz + x*y*cxy + x*z*cxz + y*z*cyz + x*y*z*cxyz);

			  real veloPreX = veloParticleX[kTimeStepOld];
			  real veloPreY = veloParticleY[kTimeStepOld];
			  real veloPreZ = veloParticleZ[kTimeStepOld];

			  real veloPostX = (veloPreX + vx1) * c1o2;
			  real veloPostY = (veloPreY + vx2) * c1o2;
			  real veloPostZ = (veloPreZ + vx3) * c1o2;

			  //real veloPostX = vx1;
			  //real veloPostY = vx2;
			  //real veloPostZ = vx3;

			  veloParticleX[kTimeStep] = veloPostX;
			  veloParticleY[kTimeStep] = veloPostY;
			  veloParticleZ[kTimeStep] = veloPostZ;
			  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  x = x + veloPostX;
			  //x = x + c1o3;
			  y = y + veloPostY;
			  z = z + veloPostZ;

			  unsigned int cbID = cellBaseID[k];
			  bool negativeDirection = false;

			  if (x >  c1o2)
			  {
				  cbID = neighborX[cbID]; 
				  x = x - c1o1;
			  }
			  if (y >  c1o2)
  			  {
				  cbID = neighborY[cbID]; 
				  y = y - c1o1;
			  }
			  if (z >  c1o2)
			  {
				  cbID = neighborZ[cbID]; 
				  z = z - c1o1;
			  }

			  real tempX = x;
			  real tempY = y;
			  real tempZ = z;

			  if ((x < -c1o2) || (y < -c1o2) || (z < -c1o2))
			  {
				  cbID = neighborWSB[cbID];
				  negativeDirection = true;
				  tempX = x + c1o1;
				  tempY = y + c1o1;
				  tempZ = z + c1o1;
			  }
			  if ((x >= -c1o2) && (negativeDirection == true))
			  {
				  cbID = neighborX[cbID]; 
				  tempX = x;
			  }
			  if ((y >= -c1o2) && (negativeDirection == true))
			  {
				  cbID = neighborY[cbID]; 
				  tempY = y;
			  }
			  if ((z >= -c1o2) && (negativeDirection == true))
			  { 
				  cbID = neighborZ[cbID]; 
				  tempZ = z;
			  }

			  x = tempX;
			  y = tempY;
			  z = tempZ;

			  localX                         = (x + c1o2) / (real)(pow((double)c2o1, (double)level));
              localY                         = (y + c1o2) / (real)(pow((double)c2o1, (double)level));
              localZ                         = (z + c1o2) / (real)(pow((double)c2o1, (double)level));
			  coordParticleXlocal[kTimeStep] = localX;
			  coordParticleYlocal[kTimeStep] = localY;
			  coordParticleZlocal[kTimeStep] = localZ;
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  real globalX = localX + coordX[cbID];
			  real globalY = localY + coordY[cbID];
			  real globalZ = localZ + coordZ[cbID];
			  coordParticleXglobal[kTimeStep] = globalX;
			  coordParticleYglobal[kTimeStep] = globalY;
			  coordParticleZglobal[kTimeStep] = globalZ;
			  //coordParticleXglobal[kTimeStep] = coordParticleXglobal[kTimeStepOld];
			  //coordParticleYglobal[kTimeStep] = coordParticleYglobal[kTimeStepOld];
			  //coordParticleZglobal[kTimeStep] = coordParticleZglobal[kTimeStepOld];
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  cellBaseID[k] = cbID;
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////	  
		}
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////






































//////////////////////////////////////////////////////////////////////////////
__global__ void MoveParticlesWithoutBCs(   real* coordX,
													  real* coordY,
													  real* coordZ, 
													  real* coordParticleXlocal,
													  real* coordParticleYlocal,
													  real* coordParticleZlocal,
													  real* coordParticleXglobal,
													  real* coordParticleYglobal,
													  real* coordParticleZglobal,
													  real* veloParticleX,
													  real* veloParticleY,
													  real* veloParticleZ,
													  real* DD,
													  real  omega,
													  unsigned int* particleID,
													  unsigned int* cellBaseID,
													  unsigned int* bcMatD,
													  unsigned int* neighborX,
													  unsigned int* neighborY,
													  unsigned int* neighborZ,
													  unsigned int* neighborWSB,
													  int level,
													  unsigned int timestep, 
													  unsigned int numberOfTimesteps, 
													  unsigned int numberOfParticles, 
													  unsigned long long numberOfLBnodes,
													  bool isEvenTimestep)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  ix = threadIdx.x;  // Globaler x-Index 
   const unsigned  iy = blockIdx.x;   // Globaler y-Index 
   const unsigned  iz = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*iz + iy) + ix;
   //////////////////////////////////////////////////////////////////////////

   //real press;
   real vx1,vx2,vx3;
   real drho_SWT,vx1_SWT,vx2_SWT,vx3_SWT;
   real drho_NWT,vx1_NWT,vx2_NWT,vx3_NWT;
   real drho_NET,vx1_NET,vx2_NET,vx3_NET;
   real drho_SET,vx1_SET,vx2_SET,vx3_SET;
   real drho_SWB,vx1_SWB,vx2_SWB,vx3_SWB;
   real drho_NWB,vx1_NWB,vx2_NWB,vx3_NWB;
   real drho_NEB,vx1_NEB,vx2_NEB,vx3_NEB;
   real drho_SEB,vx1_SEB,vx2_SEB,vx3_SEB;
   real f_E,f_W,f_N,f_S,f_T,f_B,f_NE,f_SW,f_SE,f_NW,f_TE,f_BW,f_BE,f_TW,f_TN,f_BS,f_BN,f_TS,f_ZERO,f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;
   real *feC, *fwC, *fnC, *fsC, *ftC, *fbC, *fneC, *fswC, *fseC, *fnwC, *fteC, *fbwC, *fbeC, *ftwC, *ftnC, *fbsC, *fbnC, *ftsC, *fzeroC, *ftneC, *ftswC, *ftseC, *ftnwC, *fbneC, *fbswC, *fbseC, *fbnwC;
   real kxyFromfcNEQ_SWT, kyzFromfcNEQ_SWT, kxzFromfcNEQ_SWT, kxxMyyFromfcNEQ_SWT, kxxMzzFromfcNEQ_SWT;
   real kxyFromfcNEQ_NWT, kyzFromfcNEQ_NWT, kxzFromfcNEQ_NWT, kxxMyyFromfcNEQ_NWT, kxxMzzFromfcNEQ_NWT;
   real kxyFromfcNEQ_NET, kyzFromfcNEQ_NET, kxzFromfcNEQ_NET, kxxMyyFromfcNEQ_NET, kxxMzzFromfcNEQ_NET;
   real kxyFromfcNEQ_SET, kyzFromfcNEQ_SET, kxzFromfcNEQ_SET, kxxMyyFromfcNEQ_SET, kxxMzzFromfcNEQ_SET;
   real kxyFromfcNEQ_SWB, kyzFromfcNEQ_SWB, kxzFromfcNEQ_SWB, kxxMyyFromfcNEQ_SWB, kxxMzzFromfcNEQ_SWB;
   real kxyFromfcNEQ_NWB, kyzFromfcNEQ_NWB, kxzFromfcNEQ_NWB, kxxMyyFromfcNEQ_NWB, kxxMzzFromfcNEQ_NWB;
   real kxyFromfcNEQ_NEB, kyzFromfcNEQ_NEB, kxzFromfcNEQ_NEB, kxxMyyFromfcNEQ_NEB, kxxMzzFromfcNEQ_NEB;
   real kxyFromfcNEQ_SEB, kyzFromfcNEQ_SEB, kxzFromfcNEQ_SEB, kxxMyyFromfcNEQ_SEB, kxxMzzFromfcNEQ_SEB;
   real a0, ax, ay, az, axx, ayy, azz, axy, axz, ayz, b0, bx, by, bz, bxx, byy, bzz, bxy, bxz, byz, c0, cx, cy, cz, cxx, cyy, czz, cxy, cxz, cyz, axyz, bxyz, cxyz;
   //real d0, dx, dy, dz, dxy, dxz, dyz, dxyz;

   real x,y,z;

   if(k < numberOfParticles)
   {
		/////////////////////////////////////////////////////////////
	    unsigned int kTimeStep = k + (timestep * numberOfParticles);
		/////////////////////////////////////////////////////////////
		unsigned int kCellBaseID = cellBaseID[k];
	    unsigned int BC000  = bcMatD[kCellBaseID];
	    unsigned int BCx00  = bcMatD[neighborX[kCellBaseID]];
	    unsigned int BC0y0  = bcMatD[neighborY[kCellBaseID]];
	    unsigned int BC00z  = bcMatD[neighborZ[kCellBaseID]];
	    unsigned int BCxy0  = bcMatD[neighborY[neighborX[kCellBaseID]]];
	    unsigned int BCx0z  = bcMatD[neighborZ[neighborX[kCellBaseID]]];
	    unsigned int BC0yz  = bcMatD[neighborZ[neighborY[kCellBaseID]]];
	    unsigned int BCxyz  = bcMatD[neighborZ[neighborY[neighborX[kCellBaseID]]]];
		/////////////////////////////////////////////////////////////
   		if( (BC000 == GEO_FLUID) || (BCx00 == GEO_FLUID) || (BC0y0 == GEO_FLUID) || (BC00z == GEO_FLUID) || 
			(BCxy0 == GEO_FLUID) || (BCx0z == GEO_FLUID) || (BC0yz == GEO_FLUID) || (BCxyz == GEO_FLUID) )
		{
		   if (isEvenTimestep==true)
		   {
			  feC    = &DD[dP00 * numberOfLBnodes];
			  fwC    = &DD[dM00 * numberOfLBnodes];
			  fnC    = &DD[d0P0 * numberOfLBnodes];
			  fsC    = &DD[d0M0 * numberOfLBnodes];
			  ftC    = &DD[d00P * numberOfLBnodes];
			  fbC    = &DD[d00M * numberOfLBnodes];
			  fneC   = &DD[dPP0 * numberOfLBnodes];
			  fswC   = &DD[dMM0 * numberOfLBnodes];
			  fseC   = &DD[dPM0 * numberOfLBnodes];
			  fnwC   = &DD[dMP0 * numberOfLBnodes];
			  fteC   = &DD[dP0P * numberOfLBnodes];
			  fbwC   = &DD[dM0M * numberOfLBnodes];
			  fbeC   = &DD[dP0M * numberOfLBnodes];
			  ftwC   = &DD[dM0P * numberOfLBnodes];
			  ftnC   = &DD[d0PP * numberOfLBnodes];
			  fbsC   = &DD[d0MM * numberOfLBnodes];
			  fbnC   = &DD[d0PM * numberOfLBnodes];
			  ftsC   = &DD[d0MP * numberOfLBnodes];
			  fzeroC = &DD[d000 * numberOfLBnodes];
			  ftneC  = &DD[dPPP * numberOfLBnodes];
			  ftswC  = &DD[dMMP * numberOfLBnodes];
			  ftseC  = &DD[dPMP * numberOfLBnodes];
			  ftnwC  = &DD[dMPP * numberOfLBnodes];
			  fbneC  = &DD[dPPM * numberOfLBnodes];
			  fbswC  = &DD[dMMM * numberOfLBnodes];
			  fbseC  = &DD[dPMM * numberOfLBnodes];
			  fbnwC  = &DD[dMPM * numberOfLBnodes];
		   } 			 
		   else			 
		   {			 
			  fwC    = &DD[dP00 * numberOfLBnodes];
			  feC    = &DD[dM00 * numberOfLBnodes];
			  fsC    = &DD[d0P0 * numberOfLBnodes];
			  fnC    = &DD[d0M0 * numberOfLBnodes];
			  fbC    = &DD[d00P * numberOfLBnodes];
			  ftC    = &DD[d00M * numberOfLBnodes];
			  fswC   = &DD[dPP0 * numberOfLBnodes];
			  fneC   = &DD[dMM0 * numberOfLBnodes];
			  fnwC   = &DD[dPM0 * numberOfLBnodes];
			  fseC   = &DD[dMP0 * numberOfLBnodes];
			  fbwC   = &DD[dP0P * numberOfLBnodes];
			  fteC   = &DD[dM0M * numberOfLBnodes];
			  ftwC   = &DD[dP0M * numberOfLBnodes];
			  fbeC   = &DD[dM0P * numberOfLBnodes];
			  fbsC   = &DD[d0PP * numberOfLBnodes];
			  ftnC   = &DD[d0MM * numberOfLBnodes];
			  ftsC   = &DD[d0PM * numberOfLBnodes];
			  fbnC   = &DD[d0MP * numberOfLBnodes];
			  fzeroC = &DD[d000 * numberOfLBnodes];
			  fbswC  = &DD[dPPP * numberOfLBnodes];
			  fbneC  = &DD[dMMP * numberOfLBnodes];
			  fbnwC  = &DD[dPMP * numberOfLBnodes];
			  fbseC  = &DD[dMPP * numberOfLBnodes];
			  ftswC  = &DD[dPPM * numberOfLBnodes];
			  ftneC  = &DD[dMMM * numberOfLBnodes];
			  ftnwC  = &DD[dPMM * numberOfLBnodes];
			  ftseC  = &DD[dMPM * numberOfLBnodes];
		   }

			  //////////////////////////////////////////////////////////////////////////
			  //SWB//
			  //////////////////////////////////////////////////////////////////////////
			  //index 0
			  unsigned int k0zero= cellBaseID[k];
			  unsigned int k0w   = neighborX[k0zero];
			  unsigned int k0s   = neighborY[k0zero];
			  unsigned int k0b   = neighborZ[k0zero];
			  unsigned int k0sw  = neighborY[k0w];
			  unsigned int k0bw  = neighborZ[k0w];
			  unsigned int k0bs  = neighborZ[k0s];
			  unsigned int k0bsw = neighborZ[k0sw];
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  unsigned int kzero= k0zero;
			  unsigned int kw   = k0w;   
			  unsigned int ks   = k0s;   
			  unsigned int kb   = k0b;   
			  unsigned int ksw  = k0sw;  
			  unsigned int kbw  = k0bw;  
			  unsigned int kbs  = k0bs;  
			  unsigned int kbsw = k0bsw; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_SWB = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_SWB  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_SWB);
			  vx2_SWB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_SWB);
			  vx3_SWB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_SWB);

			  kxyFromfcNEQ_SWB    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_SWB) - ((vx1_SWB*vx2_SWB)));
			  kyzFromfcNEQ_SWB    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_SWB) - ((vx2_SWB*vx3_SWB)));
			  kxzFromfcNEQ_SWB    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_SWB) - ((vx1_SWB*vx3_SWB)));
			  kxxMyyFromfcNEQ_SWB = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_SWB) - ((vx1_SWB*vx1_SWB-vx2_SWB*vx2_SWB)));
			  kxxMzzFromfcNEQ_SWB = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_SWB) - ((vx1_SWB*vx1_SWB-vx3_SWB*vx3_SWB)));

			  //////////////////////////////////////////////////////////////////////////
			  //SWT//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= kb;
			  kw   = kbw;   
			  ks   = kbs;   
			  kb   = neighborZ[kb];   
			  ksw  = kbsw;  
			  kbw  = neighborZ[kbw];  
			  kbs  = neighborZ[kbs];  
			  kbsw = neighborZ[kbsw]; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_SWT = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_SWT  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_SWT);
			  vx2_SWT  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_SWT);
			  vx3_SWT  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_SWT);

			  kxyFromfcNEQ_SWT    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_SWT) - ((vx1_SWT*vx2_SWT)));
			  kyzFromfcNEQ_SWT    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_SWT) - ((vx2_SWT*vx3_SWT)));
			  kxzFromfcNEQ_SWT    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_SWT) - ((vx1_SWT*vx3_SWT)));
			  kxxMyyFromfcNEQ_SWT = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_SWT) - ((vx1_SWT*vx1_SWT-vx2_SWT*vx2_SWT)));
			  kxxMzzFromfcNEQ_SWT = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_SWT) - ((vx1_SWT*vx1_SWT-vx3_SWT*vx3_SWT)));

			  //////////////////////////////////////////////////////////////////////////
			  //SET//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= kw;
			  kw   = neighborX[kw];   
			  ks   = ksw;   
			  kb   = kbw;   
			  ksw  = neighborX[ksw];  
			  kbw  = neighborX[kbw];  
			  kbs  = kbsw;  
			  kbsw = neighborX[kbsw]; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_SET = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_SET  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_SET);
			  vx2_SET  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_SET);
			  vx3_SET  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_SET);

			  kxyFromfcNEQ_SET    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_SET) - ((vx1_SET*vx2_SET)));
			  kyzFromfcNEQ_SET    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_SET) - ((vx2_SET*vx3_SET)));
			  kxzFromfcNEQ_SET    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_SET) - ((vx1_SET*vx3_SET)));
			  kxxMyyFromfcNEQ_SET = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_SET) - ((vx1_SET*vx1_SET-vx2_SET*vx2_SET)));
			  kxxMzzFromfcNEQ_SET = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_SET) - ((vx1_SET*vx1_SET-vx3_SET*vx3_SET)));

			  //////////////////////////////////////////////////////////////////////////
			  //SEB//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kb   = kzero;   
			  kbw  = kw;  
			  kbs  = ks;  
			  kbsw = ksw; 
			  kzero= k0w;
			  kw   = neighborX[k0w];   
			  ks   = k0sw;   
			  ksw  = neighborX[k0sw];  
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_SEB = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_SEB  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_SEB);
			  vx2_SEB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_SEB);
			  vx3_SEB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_SEB);

			  kxyFromfcNEQ_SEB    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_SEB) - ((vx1_SEB*vx2_SEB)));
			  kyzFromfcNEQ_SEB    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_SEB) - ((vx2_SEB*vx3_SEB)));
			  kxzFromfcNEQ_SEB    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_SEB) - ((vx1_SEB*vx3_SEB)));
			  kxxMyyFromfcNEQ_SEB = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_SEB) - ((vx1_SEB*vx1_SEB-vx2_SEB*vx2_SEB)));
			  kxxMzzFromfcNEQ_SEB = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_SEB) - ((vx1_SEB*vx1_SEB-vx3_SEB*vx3_SEB)));

			  //////////////////////////////////////////////////////////////////////////
			  //NWB//
			  //////////////////////////////////////////////////////////////////////////
			  //index 0
			  k0zero= k0s;
			  k0w   = k0sw;
			  k0s   = neighborY[k0s];
			  k0b   = k0bs;
			  k0sw  = neighborY[k0sw];
			  k0bw  = k0bsw;
			  k0bs  = neighborY[k0bs];
			  k0bsw = neighborY[k0bsw];
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= k0zero;
			  kw   = k0w;   
			  ks   = k0s;   
			  kb   = k0b;   
			  ksw  = k0sw;  
			  kbw  = k0bw;  
			  kbs  = k0bs;  
			  kbsw = k0bsw; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_NWB = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_NWB  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_NWB);
			  vx2_NWB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_NWB);
			  vx3_NWB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_NWB);

			  kxyFromfcNEQ_NWB    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_NWB) - ((vx1_NWB*vx2_NWB)));
			  kyzFromfcNEQ_NWB    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_NWB) - ((vx2_NWB*vx3_NWB)));
			  kxzFromfcNEQ_NWB    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_NWB) - ((vx1_NWB*vx3_NWB)));
			  kxxMyyFromfcNEQ_NWB = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_NWB) - ((vx1_NWB*vx1_NWB-vx2_NWB*vx2_NWB)));
			  kxxMzzFromfcNEQ_NWB = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_NWB) - ((vx1_NWB*vx1_NWB-vx3_NWB*vx3_NWB)));

			  //////////////////////////////////////////////////////////////////////////
			  //NWT//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= kb;
			  kw   = kbw;   
			  ks   = kbs;   
			  kb   = neighborZ[kb];   
			  ksw  = kbsw;  
			  kbw  = neighborZ[kbw];  
			  kbs  = neighborZ[kbs];  
			  kbsw = neighborZ[kbsw]; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_NWT = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_NWT  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_NWT);
			  vx2_NWT  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_NWT);
			  vx3_NWT  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_NWT);

			  kxyFromfcNEQ_NWT    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_NWT) - ((vx1_NWT*vx2_NWT)));
			  kyzFromfcNEQ_NWT    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_NWT) - ((vx2_NWT*vx3_NWT)));
			  kxzFromfcNEQ_NWT    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_NWT) - ((vx1_NWT*vx3_NWT)));
			  kxxMyyFromfcNEQ_NWT = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_NWT) - ((vx1_NWT*vx1_NWT-vx2_NWT*vx2_NWT)));
			  kxxMzzFromfcNEQ_NWT = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_NWT) - ((vx1_NWT*vx1_NWT-vx3_NWT*vx3_NWT)));

			  //////////////////////////////////////////////////////////////////////////
			  //NET//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kzero= kw;
			  kw   = neighborX[kw];   
			  ks   = ksw;   
			  kb   = kbw;   
			  ksw  = neighborX[ksw];  
			  kbw  = neighborX[kbw];  
			  kbs  = kbsw;  
			  kbsw = neighborX[kbsw]; 
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_NET = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_NET  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_NET);
			  vx2_NET  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_NET);
			  vx3_NET  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_NET);

			  kxyFromfcNEQ_NET    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_NET) - ((vx1_NET*vx2_NET)));
			  kyzFromfcNEQ_NET    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_NET) - ((vx2_NET*vx3_NET)));
			  kxzFromfcNEQ_NET    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_NET) - ((vx1_NET*vx3_NET)));
			  kxxMyyFromfcNEQ_NET = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_NET) - ((vx1_NET*vx1_NET-vx2_NET*vx2_NET)));
			  kxxMzzFromfcNEQ_NET = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_NET) - ((vx1_NET*vx1_NET-vx3_NET*vx3_NET)));

			  //////////////////////////////////////////////////////////////////////////
			  //NEB//
			  //////////////////////////////////////////////////////////////////////////
			  //index 
			  kb   = kzero;   
			  kbw  = kw;  
			  kbs  = ks;  
			  kbsw = ksw; 
			  kzero= k0w;
			  kw   = neighborX[k0w];   
			  ks   = k0sw;   
			  ksw  = neighborX[k0sw];  
			  ////////////////////////////////////////////////////////////////////////////////
			  f_E    = feC[kzero];
			  f_W    = fwC[kw];
			  f_N    = fnC[kzero];
			  f_S    = fsC[ks];
			  f_T    = ftC[kzero];
			  f_B    = fbC[kb];
			  f_NE   = fneC[kzero];
			  f_SW   = fswC[ksw];
			  f_SE   = fseC[ks];
			  f_NW   = fnwC[kw];
			  f_TE   = fteC[kzero];
			  f_BW   = fbwC[kbw];
			  f_BE   = fbeC[kb];
			  f_TW   = ftwC[kw];
			  f_TN   = ftnC[kzero];
			  f_BS   = fbsC[kbs];
			  f_BN   = fbnC[kb];
			  f_TS   = ftsC[ks];
			  f_ZERO = fzeroC[kzero];
			  f_TNE  = ftneC[kzero];
			  f_TSW  = ftswC[ksw];
			  f_TSE  = ftseC[ks];
			  f_TNW  = ftnwC[kw];
			  f_BNE  = fbneC[kb];
			  f_BSW  = fbswC[kbsw];
			  f_BSE  = fbseC[kbs];
			  f_BNW  = fbnwC[kbw];

			  drho_NEB = f_E+f_W+f_N+f_S+f_T+f_B+f_NE+f_SW+f_SE+f_NW+f_TE+f_BW+f_BE+f_TW+f_TN+f_BS+f_BN+f_TS+f_ZERO+f_TNE+f_TSW+f_TSE+f_TNW+f_BNE+f_BSW+f_BSE+f_BNW;
			  vx1_NEB  = (((f_TNE-f_BSW)+(f_TSE-f_BNW)+(f_BNE-f_TSW)+(f_BSE-f_TNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W))/(c1o1 + drho_NEB);
			  vx2_NEB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S))/(c1o1 + drho_NEB);
			  vx3_NEB  = (((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSE-f_BNW)+(f_TSW-f_BNE)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B))/(c1o1 + drho_NEB);

			  kxyFromfcNEQ_NEB    = -c3o1*omega*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho_NEB) - ((vx1_NEB*vx2_NEB)));
			  kyzFromfcNEQ_NEB    = -c3o1*omega*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho_NEB) - ((vx2_NEB*vx3_NEB)));
			  kxzFromfcNEQ_NEB    = -c3o1*omega*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho_NEB) - ((vx1_NEB*vx3_NEB)));
			  kxxMyyFromfcNEQ_NEB = -c3o2*omega *((f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE             ) / (c1o1 + drho_NEB) - ((vx1_NEB*vx1_NEB-vx2_NEB*vx2_NEB)));
			  kxxMzzFromfcNEQ_NEB = -c3o2*omega *((f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE             ) / (c1o1 + drho_NEB) - ((vx1_NEB*vx1_NEB-vx3_NEB*vx3_NEB)));

			  //////////////////////////////////////////////////////////////////////////
			  //interpolate
			  //////////////////////////////////////////////////////////////////////////
			  a0 = (-kxxMyyFromfcNEQ_NEB - kxxMyyFromfcNEQ_NET + kxxMyyFromfcNEQ_NWB + kxxMyyFromfcNEQ_NWT - 
				 kxxMyyFromfcNEQ_SEB - kxxMyyFromfcNEQ_SET + kxxMyyFromfcNEQ_SWB + kxxMyyFromfcNEQ_SWT - 
				 kxxMzzFromfcNEQ_NEB - kxxMzzFromfcNEQ_NET + kxxMzzFromfcNEQ_NWB + kxxMzzFromfcNEQ_NWT - 
				 kxxMzzFromfcNEQ_SEB - kxxMzzFromfcNEQ_SET + kxxMzzFromfcNEQ_SWB + kxxMzzFromfcNEQ_SWT - 
				 c2o1*kxyFromfcNEQ_NEB - c2o1*kxyFromfcNEQ_NET - c2o1*kxyFromfcNEQ_NWB - c2o1*kxyFromfcNEQ_NWT + 
				 c2o1*kxyFromfcNEQ_SEB + c2o1*kxyFromfcNEQ_SET + c2o1*kxyFromfcNEQ_SWB + c2o1*kxyFromfcNEQ_SWT + 
				 c2o1*kxzFromfcNEQ_NEB - c2o1*kxzFromfcNEQ_NET + c2o1*kxzFromfcNEQ_NWB - c2o1*kxzFromfcNEQ_NWT + 
				 c2o1*kxzFromfcNEQ_SEB - c2o1*kxzFromfcNEQ_SET + c2o1*kxzFromfcNEQ_SWB - c2o1*kxzFromfcNEQ_SWT + 
				 c8o1*vx1_NEB + c8o1*vx1_NET + c8o1*vx1_NWB + c8o1*vx1_NWT + c8o1*vx1_SEB + 
				 c8o1*vx1_SET + c8o1*vx1_SWB + c8o1*vx1_SWT + c2o1*vx2_NEB + c2o1*vx2_NET - 
				 c2o1*vx2_NWB - c2o1*vx2_NWT - c2o1*vx2_SEB - c2o1*vx2_SET + c2o1*vx2_SWB + 
				 c2o1*vx2_SWT - c2o1*vx3_NEB + c2o1*vx3_NET + c2o1*vx3_NWB - c2o1*vx3_NWT - 
				 c2o1*vx3_SEB + c2o1*vx3_SET + c2o1*vx3_SWB - c2o1*vx3_SWT)/c64o1;
			  b0 = (c2o1*kxxMyyFromfcNEQ_NEB + c2o1*kxxMyyFromfcNEQ_NET + c2o1*kxxMyyFromfcNEQ_NWB + c2o1*kxxMyyFromfcNEQ_NWT - 
				 c2o1*kxxMyyFromfcNEQ_SEB - c2o1*kxxMyyFromfcNEQ_SET - c2o1*kxxMyyFromfcNEQ_SWB - c2o1*kxxMyyFromfcNEQ_SWT - 
				 kxxMzzFromfcNEQ_NEB - kxxMzzFromfcNEQ_NET - kxxMzzFromfcNEQ_NWB - kxxMzzFromfcNEQ_NWT + 
				 kxxMzzFromfcNEQ_SEB + kxxMzzFromfcNEQ_SET + kxxMzzFromfcNEQ_SWB + kxxMzzFromfcNEQ_SWT - 
				 c2o1*kxyFromfcNEQ_NEB - c2o1*kxyFromfcNEQ_NET + c2o1*kxyFromfcNEQ_NWB + c2o1*kxyFromfcNEQ_NWT - 
				 c2o1*kxyFromfcNEQ_SEB - c2o1*kxyFromfcNEQ_SET + c2o1*kxyFromfcNEQ_SWB + c2o1*kxyFromfcNEQ_SWT + 
				 c2o1*kyzFromfcNEQ_NEB - c2o1*kyzFromfcNEQ_NET + c2o1*kyzFromfcNEQ_NWB - c2o1*kyzFromfcNEQ_NWT + 
				 c2o1*kyzFromfcNEQ_SEB - c2o1*kyzFromfcNEQ_SET + c2o1*kyzFromfcNEQ_SWB - c2o1*kyzFromfcNEQ_SWT + 
				 c2o1*vx1_NEB + c2o1*vx1_NET - c2o1*vx1_NWB - c2o1*vx1_NWT - 
				 c2o1*vx1_SEB - c2o1*vx1_SET + c2o1*vx1_SWB + c2o1*vx1_SWT + 
				 c8o1*vx2_NEB + c8o1*vx2_NET + c8o1*vx2_NWB + c8o1*vx2_NWT + 
				 c8o1*vx2_SEB + c8o1*vx2_SET + c8o1*vx2_SWB + c8o1*vx2_SWT - 
				 c2o1*vx3_NEB + c2o1*vx3_NET - c2o1*vx3_NWB + c2o1*vx3_NWT + 
				 c2o1*vx3_SEB - c2o1*vx3_SET + c2o1*vx3_SWB - c2o1*vx3_SWT)/c64o1;

			  //b0 = ((eight*vx2_NEB + eight*vx2_SWT) + (eight*vx2_NET + eight*vx2_SWB) + (eight*vx2_NWB + eight*vx2_SET) + (eight*vx2_NWT + eight*vx2_SEB))/sixtyfour;

			  c0 = (kxxMyyFromfcNEQ_NEB - kxxMyyFromfcNEQ_NET + kxxMyyFromfcNEQ_NWB - kxxMyyFromfcNEQ_NWT + 
				 kxxMyyFromfcNEQ_SEB - kxxMyyFromfcNEQ_SET + kxxMyyFromfcNEQ_SWB - kxxMyyFromfcNEQ_SWT - 
				 c2o1*kxxMzzFromfcNEQ_NEB + c2o1*kxxMzzFromfcNEQ_NET - c2o1*kxxMzzFromfcNEQ_NWB + c2o1*kxxMzzFromfcNEQ_NWT - 
				 c2o1*kxxMzzFromfcNEQ_SEB + c2o1*kxxMzzFromfcNEQ_SET - c2o1*kxxMzzFromfcNEQ_SWB + c2o1*kxxMzzFromfcNEQ_SWT - 
				 c2o1*kxzFromfcNEQ_NEB - c2o1*kxzFromfcNEQ_NET + c2o1*kxzFromfcNEQ_NWB + c2o1*kxzFromfcNEQ_NWT - 
				 c2o1*kxzFromfcNEQ_SEB - c2o1*kxzFromfcNEQ_SET + c2o1*kxzFromfcNEQ_SWB + c2o1*kxzFromfcNEQ_SWT - 
				 c2o1*kyzFromfcNEQ_NEB - c2o1*kyzFromfcNEQ_NET - c2o1*kyzFromfcNEQ_NWB - c2o1*kyzFromfcNEQ_NWT + 
				 c2o1*kyzFromfcNEQ_SEB + c2o1*kyzFromfcNEQ_SET + c2o1*kyzFromfcNEQ_SWB + c2o1*kyzFromfcNEQ_SWT - 
				 c2o1*vx1_NEB + c2o1*vx1_NET + c2o1*vx1_NWB - c2o1*vx1_NWT - 
				 c2o1*vx1_SEB + c2o1*vx1_SET + c2o1*vx1_SWB - c2o1*vx1_SWT - 
				 c2o1*vx2_NEB + c2o1*vx2_NET - c2o1*vx2_NWB + c2o1*vx2_NWT + 
				 c2o1*vx2_SEB - c2o1*vx2_SET + c2o1*vx2_SWB - c2o1*vx2_SWT + 
				 c8o1*vx3_NEB + c8o1*vx3_NET + c8o1*vx3_NWB + c8o1*vx3_NWT + 
				 c8o1*vx3_SEB + c8o1*vx3_SET + c8o1*vx3_SWB + c8o1*vx3_SWT)/c64o1;
			  ax = (vx1_NEB + vx1_NET - vx1_NWB - vx1_NWT + vx1_SEB + vx1_SET - vx1_SWB - vx1_SWT)/c4o1;
			  bx = (vx2_NEB + vx2_NET - vx2_NWB - vx2_NWT + vx2_SEB + vx2_SET - vx2_SWB - vx2_SWT)/c4o1;
			  //bx = ((vx2_NEB - vx2_SWT) + (vx2_NET - vx2_SWB) + (vx2_SET - vx2_NWB) + (vx2_SEB - vx2_NWT))/four;
			  cx = (vx3_NEB + vx3_NET - vx3_NWB - vx3_NWT + vx3_SEB + vx3_SET - vx3_SWB - vx3_SWT)/c4o1;
			  axx= (kxxMyyFromfcNEQ_NEB + kxxMyyFromfcNEQ_NET - kxxMyyFromfcNEQ_NWB - kxxMyyFromfcNEQ_NWT + 
				 kxxMyyFromfcNEQ_SEB + kxxMyyFromfcNEQ_SET - kxxMyyFromfcNEQ_SWB - kxxMyyFromfcNEQ_SWT + 
				 kxxMzzFromfcNEQ_NEB + kxxMzzFromfcNEQ_NET - kxxMzzFromfcNEQ_NWB - kxxMzzFromfcNEQ_NWT + 
				 kxxMzzFromfcNEQ_SEB + kxxMzzFromfcNEQ_SET - kxxMzzFromfcNEQ_SWB - kxxMzzFromfcNEQ_SWT + 
				 c2o1*vx2_NEB + c2o1*vx2_NET - c2o1*vx2_NWB - c2o1*vx2_NWT - 
				 c2o1*vx2_SEB - c2o1*vx2_SET + c2o1*vx2_SWB + c2o1*vx2_SWT - 
				 c2o1*vx3_NEB + c2o1*vx3_NET + c2o1*vx3_NWB - c2o1*vx3_NWT - 
				 c2o1*vx3_SEB + c2o1*vx3_SET + c2o1*vx3_SWB - c2o1*vx3_SWT)/c16o1;
			  bxx= (kxyFromfcNEQ_NEB + kxyFromfcNEQ_NET - kxyFromfcNEQ_NWB - kxyFromfcNEQ_NWT + 
				 kxyFromfcNEQ_SEB + kxyFromfcNEQ_SET - kxyFromfcNEQ_SWB - kxyFromfcNEQ_SWT - 
				 c2o1*vx1_NEB - c2o1*vx1_NET + c2o1*vx1_NWB + c2o1*vx1_NWT + 
				 c2o1*vx1_SEB + c2o1*vx1_SET - c2o1*vx1_SWB - c2o1*vx1_SWT)/c8o1;
			  cxx= (kxzFromfcNEQ_NEB + kxzFromfcNEQ_NET - kxzFromfcNEQ_NWB - kxzFromfcNEQ_NWT + 
				 kxzFromfcNEQ_SEB + kxzFromfcNEQ_SET - kxzFromfcNEQ_SWB - kxzFromfcNEQ_SWT + 
				 c2o1*vx1_NEB - c2o1*vx1_NET - c2o1*vx1_NWB + c2o1*vx1_NWT + 
				 c2o1*vx1_SEB - c2o1*vx1_SET - c2o1*vx1_SWB + c2o1*vx1_SWT)/c8o1;
			  ay = (vx1_NEB + vx1_NET + vx1_NWB + vx1_NWT - vx1_SEB - vx1_SET - vx1_SWB - vx1_SWT)/c4o1;
			  by = (vx2_NEB + vx2_NET + vx2_NWB + vx2_NWT - vx2_SEB - vx2_SET - vx2_SWB - vx2_SWT)/c4o1;
			  cy = (vx3_NEB + vx3_NET + vx3_NWB + vx3_NWT - vx3_SEB - vx3_SET - vx3_SWB - vx3_SWT)/c4o1;
			  ayy= (kxyFromfcNEQ_NEB + kxyFromfcNEQ_NET + kxyFromfcNEQ_NWB + kxyFromfcNEQ_NWT - 
				 kxyFromfcNEQ_SEB - kxyFromfcNEQ_SET - kxyFromfcNEQ_SWB - kxyFromfcNEQ_SWT - 
				 c2o1*vx2_NEB - c2o1*vx2_NET + c2o1*vx2_NWB + c2o1*vx2_NWT + 
				 c2o1*vx2_SEB + c2o1*vx2_SET - c2o1*vx2_SWB - c2o1*vx2_SWT)/c8o1;
			  byy= (-c2o1*kxxMyyFromfcNEQ_NEB - c2o1*kxxMyyFromfcNEQ_NET - c2o1*kxxMyyFromfcNEQ_NWB - c2o1*kxxMyyFromfcNEQ_NWT + 
				 c2o1*kxxMyyFromfcNEQ_SEB + c2o1*kxxMyyFromfcNEQ_SET + c2o1*kxxMyyFromfcNEQ_SWB + c2o1*kxxMyyFromfcNEQ_SWT + 
				 kxxMzzFromfcNEQ_NEB + kxxMzzFromfcNEQ_NET + kxxMzzFromfcNEQ_NWB + kxxMzzFromfcNEQ_NWT - 
				 kxxMzzFromfcNEQ_SEB - kxxMzzFromfcNEQ_SET - kxxMzzFromfcNEQ_SWB - kxxMzzFromfcNEQ_SWT + 
				 c2o1*vx1_NEB + c2o1*vx1_NET - c2o1*vx1_NWB - c2o1*vx1_NWT - 
				 c2o1*vx1_SEB - c2o1*vx1_SET + c2o1*vx1_SWB + c2o1*vx1_SWT - 
				 c2o1*vx3_NEB + c2o1*vx3_NET - c2o1*vx3_NWB + c2o1*vx3_NWT + 
				 c2o1*vx3_SEB - c2o1*vx3_SET + c2o1*vx3_SWB - c2o1*vx3_SWT)/c16o1;
			  cyy= (kyzFromfcNEQ_NEB + kyzFromfcNEQ_NET + kyzFromfcNEQ_NWB + kyzFromfcNEQ_NWT - 
				 kyzFromfcNEQ_SEB - kyzFromfcNEQ_SET - kyzFromfcNEQ_SWB - kyzFromfcNEQ_SWT + 
				 c2o1*vx2_NEB - c2o1*vx2_NET + c2o1*vx2_NWB - c2o1*vx2_NWT - 
				 c2o1*vx2_SEB + c2o1*vx2_SET - c2o1*vx2_SWB + c2o1*vx2_SWT)/c8o1;
			  az = (-vx1_NEB + vx1_NET - vx1_NWB + vx1_NWT - vx1_SEB + vx1_SET - vx1_SWB + vx1_SWT)/c4o1;
			  //bz = (-vx2_NEB + vx2_NET - vx2_NWB + vx2_NWT - vx2_SEB + vx2_SET - vx2_SWB + vx2_SWT)/four;
			  bz = ((vx2_SWT - vx2_NEB) + (vx2_NET - vx2_SWB) + (vx2_SET - vx2_NWB) + (vx2_NWT - vx2_SEB))/c4o1;
			  cz = (-vx3_NEB + vx3_NET - vx3_NWB + vx3_NWT - vx3_SEB + vx3_SET - vx3_SWB + vx3_SWT)/c4o1;
			  azz= (-kxzFromfcNEQ_NEB + kxzFromfcNEQ_NET - kxzFromfcNEQ_NWB + kxzFromfcNEQ_NWT - 
				 kxzFromfcNEQ_SEB + kxzFromfcNEQ_SET - kxzFromfcNEQ_SWB + kxzFromfcNEQ_SWT + 
				 c2o1*vx3_NEB - c2o1*vx3_NET - c2o1*vx3_NWB + c2o1*vx3_NWT + 
				 c2o1*vx3_SEB - c2o1*vx3_SET - c2o1*vx3_SWB + c2o1*vx3_SWT)/c8o1;
			  bzz= (-kyzFromfcNEQ_NEB + kyzFromfcNEQ_NET - kyzFromfcNEQ_NWB + kyzFromfcNEQ_NWT - 
				 kyzFromfcNEQ_SEB + kyzFromfcNEQ_SET - kyzFromfcNEQ_SWB + kyzFromfcNEQ_SWT + 
				 c2o1*vx3_NEB - c2o1*vx3_NET + c2o1*vx3_NWB - c2o1*vx3_NWT - 
				 c2o1*vx3_SEB + c2o1*vx3_SET - c2o1*vx3_SWB + c2o1*vx3_SWT)/c8o1;
			  czz= (-kxxMyyFromfcNEQ_NEB + kxxMyyFromfcNEQ_NET - kxxMyyFromfcNEQ_NWB + kxxMyyFromfcNEQ_NWT - 
				 kxxMyyFromfcNEQ_SEB + kxxMyyFromfcNEQ_SET - kxxMyyFromfcNEQ_SWB + kxxMyyFromfcNEQ_SWT + 
				 c2o1*kxxMzzFromfcNEQ_NEB - c2o1*kxxMzzFromfcNEQ_NET + c2o1*kxxMzzFromfcNEQ_NWB - c2o1*kxxMzzFromfcNEQ_NWT + 
				 c2o1*kxxMzzFromfcNEQ_SEB - c2o1*kxxMzzFromfcNEQ_SET + c2o1*kxxMzzFromfcNEQ_SWB - c2o1*kxxMzzFromfcNEQ_SWT - 
				 c2o1*vx1_NEB + c2o1*vx1_NET + c2o1*vx1_NWB - c2o1*vx1_NWT - 
				 c2o1*vx1_SEB + c2o1*vx1_SET + c2o1*vx1_SWB - c2o1*vx1_SWT - 
				 c2o1*vx2_NEB + c2o1*vx2_NET - c2o1*vx2_NWB + c2o1*vx2_NWT + 
				 c2o1*vx2_SEB - c2o1*vx2_SET + c2o1*vx2_SWB - c2o1*vx2_SWT)/c16o1;
			  axy= (vx1_NEB + vx1_NET - vx1_NWB - vx1_NWT - vx1_SEB - vx1_SET + vx1_SWB + vx1_SWT)/c2o1;
			  bxy= (vx2_NEB + vx2_NET - vx2_NWB - vx2_NWT - vx2_SEB - vx2_SET + vx2_SWB + vx2_SWT)/c2o1;
			  cxy= (vx3_NEB + vx3_NET - vx3_NWB - vx3_NWT - vx3_SEB - vx3_SET + vx3_SWB + vx3_SWT)/c2o1;
			  axz= (-vx1_NEB + vx1_NET + vx1_NWB - vx1_NWT - vx1_SEB + vx1_SET + vx1_SWB - vx1_SWT)/c2o1;
			  bxz= (-vx2_NEB + vx2_NET + vx2_NWB - vx2_NWT - vx2_SEB + vx2_SET + vx2_SWB - vx2_SWT)/c2o1;
			  cxz= (-vx3_NEB + vx3_NET + vx3_NWB - vx3_NWT - vx3_SEB + vx3_SET + vx3_SWB - vx3_SWT)/c2o1;
			  ayz= (-vx1_NEB + vx1_NET - vx1_NWB + vx1_NWT + vx1_SEB - vx1_SET + vx1_SWB - vx1_SWT)/c2o1;
			  byz= (-vx2_NEB + vx2_NET - vx2_NWB + vx2_NWT + vx2_SEB - vx2_SET + vx2_SWB - vx2_SWT)/c2o1;
			  cyz= (-vx3_NEB + vx3_NET - vx3_NWB + vx3_NWT + vx3_SEB - vx3_SET + vx3_SWB - vx3_SWT)/c2o1;
			  axyz=-vx1_NEB + vx1_NET + vx1_NWB - vx1_NWT + vx1_SEB - vx1_SET - vx1_SWB + vx1_SWT;
			  bxyz=-vx2_NEB + vx2_NET + vx2_NWB - vx2_NWT + vx2_SEB - vx2_SET - vx2_SWB + vx2_SWT;
			  cxyz=-vx3_NEB + vx3_NET + vx3_NWB - vx3_NWT + vx3_SEB - vx3_SET - vx3_SWB + vx3_SWT;
			  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  //drho
			//   d0   = ( drho_NEB + drho_NET + drho_NWB + drho_NWT + drho_SEB + drho_SET + drho_SWB + drho_SWT) * c1o8;
			//   dx   = ( drho_NEB + drho_NET - drho_NWB - drho_NWT + drho_SEB + drho_SET - drho_SWB - drho_SWT) * c1o4;
			//   dy   = ( drho_NEB + drho_NET + drho_NWB + drho_NWT - drho_SEB - drho_SET - drho_SWB - drho_SWT) * c1o4;
			//   dz   = (-drho_NEB + drho_NET - drho_NWB + drho_NWT - drho_SEB + drho_SET - drho_SWB + drho_SWT) * c1o4;
			//   dxy  = ( drho_NEB + drho_NET - drho_NWB - drho_NWT - drho_SEB - drho_SET + drho_SWB + drho_SWT) * c1o2;
			//   dxz  = (-drho_NEB + drho_NET + drho_NWB - drho_NWT - drho_SEB + drho_SET + drho_SWB - drho_SWT) * c1o2;
			//   dyz  = (-drho_NEB + drho_NET - drho_NWB + drho_NWT + drho_SEB - drho_SET + drho_SWB - drho_SWT) * c1o2;
			//   dxyz =  -drho_NEB + drho_NET + drho_NWB - drho_NWT + drho_SEB - drho_SET - drho_SWB + drho_SWT;
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  unsigned int kTimeStepOld = kTimeStep - numberOfParticles;
			  real localX = coordParticleXlocal[kTimeStepOld];
			  real localY = coordParticleYlocal[kTimeStepOld];
			  real localZ = coordParticleZlocal[kTimeStepOld];

			  x = (localX * (real)(pow((double)c2o1, (double)level))) - c1o2; //-c1o4;
              y = (localY * (real)(pow((double)c2o1, (double)level))) - c1o2; //-c1o4;
              z = (localZ * (real)(pow((double)c2o1, (double)level))) - c1o2; //-c1o4;
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  //press = d0 + x*dx + y*dy + z*dz + x*y*dxy + x*z*dxz + y*z*dyz + x*y*z*dxyz;
			  vx1 = (a0 + x*ax + y*ay + z*az + x*x*axx + y*y*ayy + z*z*azz + x*y*axy + x*z*axz + y*z*ayz + x*y*z*axyz);
			  vx2 = (b0 + x*bx + y*by + z*bz + x*x*bxx + y*y*byy + z*z*bzz + x*y*bxy + x*z*bxz + y*z*byz + x*y*z*bxyz);
			  vx3 = (c0 + x*cx + y*cy + z*cz + x*x*cxx + y*y*cyy + z*z*czz + x*y*cxy + x*z*cxz + y*z*cyz + x*y*z*cxyz);

			  real veloPreX = veloParticleX[kTimeStepOld];
			  real veloPreY = veloParticleY[kTimeStepOld];
			  real veloPreZ = veloParticleZ[kTimeStepOld];

			  real veloPostX = (veloPreX + vx1) * c1o2;
			  real veloPostY = (veloPreY + vx2) * c1o2;
			  real veloPostZ = (veloPreZ + vx3) * c1o2;

			  //real veloPostX = vx1;
			  //real veloPostY = vx2;
			  //real veloPostZ = vx3;

			  veloParticleX[kTimeStep] = veloPostX;
			  veloParticleY[kTimeStep] = veloPostY;
			  veloParticleZ[kTimeStep] = veloPostZ;
			  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  x = x + veloPostX;
			  //x = x + c1o3;
			  y = y + veloPostY;
			  z = z + veloPostZ;

			  unsigned int cbID = cellBaseID[k];
			  bool negativeDirection = false;

			  if (x >  c1o2)
			  {
				  cbID = neighborX[cbID]; 
				  x = x - c1o1;
			  }
			  if (y >  c1o2)
  			  {
				  cbID = neighborY[cbID]; 
				  y = y - c1o1;
			  }
			  if (z >  c1o2)
			  {
				  cbID = neighborZ[cbID]; 
				  z = z - c1o1;
			  }

			  real tempX = x;
			  real tempY = y;
			  real tempZ = z;

			  if ((x < -c1o2) || (y < -c1o2) || (z < -c1o2))
			  {
				  cbID = neighborWSB[cbID];
				  negativeDirection = true;
				  tempX = x + c1o1;
				  tempY = y + c1o1;
				  tempZ = z + c1o1;
			  }
			  if ((x >= -c1o2) && (negativeDirection == true))
			  {
				  cbID = neighborX[cbID]; 
				  tempX = x;
			  }
			  if ((y >= -c1o2) && (negativeDirection == true))
			  {
				  cbID = neighborY[cbID]; 
				  tempY = y;
			  }
			  if ((z >= -c1o2) && (negativeDirection == true))
			  { 
				  cbID = neighborZ[cbID]; 
				  tempZ = z;
			  }

			  x = tempX;
			  y = tempY;
			  z = tempZ;

			  localX                         = (x + c1o2) / (real)(pow((double)c2o1, (double)level));
              localY                         = (y + c1o2) / (real)(pow((double)c2o1, (double)level));
              localZ                         = (z + c1o2) / (real)(pow((double)c2o1, (double)level));
			  coordParticleXlocal[kTimeStep] = localX;
			  coordParticleYlocal[kTimeStep] = localY;
			  coordParticleZlocal[kTimeStep] = localZ;
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  real globalX = localX + coordX[cbID];
			  real globalY = localY + coordY[cbID];
			  real globalZ = localZ + coordZ[cbID];
			  coordParticleXglobal[kTimeStep] = globalX;
			  coordParticleYglobal[kTimeStep] = globalY;
			  coordParticleZglobal[kTimeStep] = globalZ;
			  //coordParticleXglobal[kTimeStep] = coordParticleXglobal[kTimeStepOld];
			  //coordParticleYglobal[kTimeStep] = coordParticleYglobal[kTimeStepOld];
			  //coordParticleZglobal[kTimeStep] = coordParticleZglobal[kTimeStepOld];
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			  cellBaseID[k] = cbID;
			  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////	  
		}
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////






































//////////////////////////////////////////////////////////////////////////////
__global__ void ParticleNoSlipDeviceComp27(real* coordX,
													  real* coordY,
													  real* coordZ, 
													  real* coordParticleXlocal,
													  real* coordParticleYlocal,
													  real* coordParticleZlocal,
													  real* coordParticleXglobal,
													  real* coordParticleYglobal,
													  real* coordParticleZglobal,
													  real* veloParticleX,
													  real* veloParticleY,
													  real* veloParticleZ,
													  real* randArray,
													  real* DD,
													  real  omega,
													  unsigned int* particleID,
													  unsigned int* cellBaseID,
													  unsigned int* bcMatD,
													  unsigned int* neighborX,
													  unsigned int* neighborY,
													  unsigned int* neighborZ,
													  unsigned int* neighborWSB,
													  int level,
													  unsigned int numberOfTimesteps, 
													  unsigned int timestep, 
													  unsigned int numberOfParticles, 
													  int* k_Q, 
													  real* QQ,
													  unsigned int  numberOfBCnodes,
													  real* NormalX,
													  real* NormalY,
													  real* NormalZ,
													  unsigned long long numberOfLBnodes, 
													  bool isEvenTimestep)
{

	//TODO: What is this function for???

   //Distributions27 D;
   //if (isEvenTimestep==true)
   //{
   //   D.f[dP00] = &DD[dP00 * size_Mat];
   //   D.f[dM00] = &DD[dM00 * size_Mat];
   //   D.f[d0P0] = &DD[d0P0 * size_Mat];
   //   D.f[d0M0] = &DD[d0M0 * size_Mat];
   //   D.f[d00P] = &DD[d00P * size_Mat];
   //   D.f[d00M] = &DD[d00M * size_Mat];
   //   D.f[dPP0] = &DD[dPP0 * size_Mat];
   //   D.f[dMM0] = &DD[dMM0 * size_Mat];
   //   D.f[dPM0] = &DD[dPM0 * size_Mat];
   //   D.f[dMP0] = &DD[dMP0 * size_Mat];
   //   D.f[dP0P] = &DD[dP0P * size_Mat];
   //   D.f[dM0M] = &DD[dM0M * size_Mat];
   //   D.f[dP0M] = &DD[dP0M * size_Mat];
   //   D.f[dM0P] = &DD[dM0P * size_Mat];
   //   D.f[d0PP] = &DD[d0PP * size_Mat];
   //   D.f[d0MM] = &DD[d0MM * size_Mat];
   //   D.f[d0PM] = &DD[d0PM * size_Mat];
   //   D.f[d0MP] = &DD[d0MP * size_Mat];
   //   D.f[d000] = &DD[d000 * size_Mat];
   //   D.f[dPPP] = &DD[dPPP * size_Mat];
   //   D.f[dMMP] = &DD[dMMP * size_Mat];
   //   D.f[dPMP] = &DD[dPMP * size_Mat];
   //   D.f[dMPP] = &DD[dMPP * size_Mat];
   //   D.f[dPPM] = &DD[dPPM * size_Mat];
   //   D.f[dMMM] = &DD[dMMM * size_Mat];
   //   D.f[dPMM] = &DD[dPMM * size_Mat];
   //   D.f[dMPM] = &DD[dMPM * size_Mat];
   //} 
   //else
   //{
   //   D.f[dM00] = &DD[dP00 * size_Mat];
   //   D.f[dP00] = &DD[dM00 * size_Mat];
   //   D.f[d0M0] = &DD[d0P0 * size_Mat];
   //   D.f[d0P0] = &DD[d0M0 * size_Mat];
   //   D.f[d00M] = &DD[d00P * size_Mat];
   //   D.f[d00P] = &DD[d00M * size_Mat];
   //   D.f[dMM0] = &DD[dPP0 * size_Mat];
   //   D.f[dPP0] = &DD[dMM0 * size_Mat];
   //   D.f[dMP0] = &DD[dPM0 * size_Mat];
   //   D.f[dPM0] = &DD[dMP0 * size_Mat];
   //   D.f[dM0M] = &DD[dP0P * size_Mat];
   //   D.f[dP0P] = &DD[dM0M * size_Mat];
   //   D.f[dM0P] = &DD[dP0M * size_Mat];
   //   D.f[dP0M] = &DD[dM0P * size_Mat];
   //   D.f[d0MM] = &DD[d0PP * size_Mat];
   //   D.f[d0PP] = &DD[d0MM * size_Mat];
   //   D.f[d0MP] = &DD[d0PM * size_Mat];
   //   D.f[d0PM] = &DD[d0MP * size_Mat];
   //   D.f[d000] = &DD[d000 * size_Mat];
   //   D.f[dPPP] = &DD[dMMM * size_Mat];
   //   D.f[dMMP] = &DD[dPPM * size_Mat];
   //   D.f[dPMP] = &DD[dMPM * size_Mat];
   //   D.f[dMPP] = &DD[dPMM * size_Mat];
   //   D.f[dPPM] = &DD[dMMP * size_Mat];
   //   D.f[dMMM] = &DD[dPPP * size_Mat];
   //   D.f[dPMM] = &DD[dMPP * size_Mat];
   //   D.f[dMPM] = &DD[dPMP * size_Mat];
   //}
   //////////////////////////////////////////////////////////////////////////////////
   //const unsigned  x = threadIdx.x;  // Globaler x-Index 
   //const unsigned  y = blockIdx.x;   // Globaler y-Index 
   //const unsigned  z = blockIdx.y;   // Globaler z-Index 

   //const unsigned nx = blockDim.x;
   //const unsigned ny = gridDim.x;

   //const unsigned k = nx*(ny*z + y) + x;
   ////////////////////////////////////////////////////////////////////////////

   //if(k <  numberOfBCnodes)
   //{
   //   ////////////////////////////////////////////////////////////////////////////////
   //   real *q_dirW, *q_dirS, *q_dirB;
   // //   real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
   // //         *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
   // //         *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
   // //         *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
   // //         *q_dirBSE, *q_dirBNW; 
   // //   q_dirE   = &QQ[dP00 * numberOfBCnodes];
   //    q_dirW   = &QQ[dM00 * numberOfBCnodes];
   // //   q_dirN   = &QQ[d0P0 * numberOfBCnodes];
   //    q_dirS   = &QQ[d0M0 * numberOfBCnodes];
   // //   q_dirT   = &QQ[d00P * numberOfBCnodes];
   //    q_dirB   = &QQ[d00M * numberOfBCnodes];
   // //   q_dirNE  = &QQ[dPP0 * numberOfBCnodes];
   // //   q_dirSW  = &QQ[dMM0 * numberOfBCnodes];
   // //   q_dirSE  = &QQ[dPM0 * numberOfBCnodes];
   // //   q_dirNW  = &QQ[dMP0 * numberOfBCnodes];
   // //   q_dirTE  = &QQ[dP0P * numberOfBCnodes];
   // //   q_dirBW  = &QQ[dM0M * numberOfBCnodes];
   // //   q_dirBE  = &QQ[dP0M * numberOfBCnodes];
   // //   q_dirTW  = &QQ[dM0P * numberOfBCnodes];
   // //   q_dirTN  = &QQ[d0PP * numberOfBCnodes];
   // //   q_dirBS  = &QQ[d0MM * numberOfBCnodes];
   // //   q_dirBN  = &QQ[d0PM * numberOfBCnodes];
   // //   q_dirTS  = &QQ[d0MP * numberOfBCnodes];
   // //   q_dirTNE = &QQ[dPPP * numberOfBCnodes];
   // //   q_dirTSW = &QQ[dMMP * numberOfBCnodes];
   // //   q_dirTSE = &QQ[dPMP * numberOfBCnodes];
   // //   q_dirTNW = &QQ[dMPP * numberOfBCnodes];
   // //   q_dirBNE = &QQ[dPPM * numberOfBCnodes];
   // //   q_dirBSW = &QQ[dMMM * numberOfBCnodes];
   // //   q_dirBSE = &QQ[dPMM * numberOfBCnodes];
   // //   q_dirBNW = &QQ[dMPM * numberOfBCnodes];
   //   ////////////////////////////////////////////////////////////////////////////////
   // //   real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
   // //           *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
   // //           *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
   // //           *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
   // //           *nx_dirBSE, *nx_dirBNW; 
   // //   nx_dirE   = &NormalX[dP00 * numberOfBCnodes];
   // //   nx_dirW   = &NormalX[dM00 * numberOfBCnodes];
   // //   nx_dirN   = &NormalX[d0P0 * numberOfBCnodes];
   // //   nx_dirS   = &NormalX[d0M0 * numberOfBCnodes];
   // //   nx_dirT   = &NormalX[d00P * numberOfBCnodes];
   // //   nx_dirB   = &NormalX[d00M * numberOfBCnodes];
   // //   nx_dirNE  = &NormalX[dPP0 * numberOfBCnodes];
   // //   nx_dirSW  = &NormalX[dMM0 * numberOfBCnodes];
   // //   nx_dirSE  = &NormalX[dPM0 * numberOfBCnodes];
   // //   nx_dirNW  = &NormalX[dMP0 * numberOfBCnodes];
   // //   nx_dirTE  = &NormalX[dP0P * numberOfBCnodes];
   // //   nx_dirBW  = &NormalX[dM0M * numberOfBCnodes];
   // //   nx_dirBE  = &NormalX[dP0M * numberOfBCnodes];
   // //   nx_dirTW  = &NormalX[dM0P * numberOfBCnodes];
   // //   nx_dirTN  = &NormalX[d0PP * numberOfBCnodes];
   // //   nx_dirBS  = &NormalX[d0MM * numberOfBCnodes];
   // //   nx_dirBN  = &NormalX[d0PM * numberOfBCnodes];
   // //   nx_dirTS  = &NormalX[d0MP * numberOfBCnodes];
   // //   nx_dirTNE = &NormalX[dPPP * numberOfBCnodes];
   // //   nx_dirTSW = &NormalX[dMMP * numberOfBCnodes];
   // //   nx_dirTSE = &NormalX[dPMP * numberOfBCnodes];
   // //   nx_dirTNW = &NormalX[dMPP * numberOfBCnodes];
   // //   nx_dirBNE = &NormalX[dPPM * numberOfBCnodes];
   // //   nx_dirBSW = &NormalX[dMMM * numberOfBCnodes];
   // //   nx_dirBSE = &NormalX[dPMM * numberOfBCnodes];
   // //   nx_dirBNW = &NormalX[dMPM * numberOfBCnodes];
   //   ////////////////////////////////////////////////////////////////////////////////
   // //   real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
   // //           *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
   // //           *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
   // //           *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
   // //           *ny_dirBSE, *ny_dirBNW; 
   // //   ny_dirE   = &NormalY[dP00 * numberOfBCnodes];
   // //   ny_dirW   = &NormalY[dM00 * numberOfBCnodes];
   // //   ny_dirN   = &NormalY[d0P0 * numberOfBCnodes];
   // //   ny_dirS   = &NormalY[d0M0 * numberOfBCnodes];
   // //   ny_dirT   = &NormalY[d00P * numberOfBCnodes];
   // //   ny_dirB   = &NormalY[d00M * numberOfBCnodes];
   // //   ny_dirNE  = &NormalY[dPP0 * numberOfBCnodes];
   // //   ny_dirSW  = &NormalY[dMM0 * numberOfBCnodes];
   // //   ny_dirSE  = &NormalY[dPM0 * numberOfBCnodes];
   // //   ny_dirNW  = &NormalY[dMP0 * numberOfBCnodes];
   // //   ny_dirTE  = &NormalY[dP0P * numberOfBCnodes];
   // //   ny_dirBW  = &NormalY[dM0M * numberOfBCnodes];
   // //   ny_dirBE  = &NormalY[dP0M * numberOfBCnodes];
   // //   ny_dirTW  = &NormalY[dM0P * numberOfBCnodes];
   // //   ny_dirTN  = &NormalY[d0PP * numberOfBCnodes];
   // //   ny_dirBS  = &NormalY[d0MM * numberOfBCnodes];
   // //   ny_dirBN  = &NormalY[d0PM * numberOfBCnodes];
   // //   ny_dirTS  = &NormalY[d0MP * numberOfBCnodes];
   // //   ny_dirTNE = &NormalY[dPPP * numberOfBCnodes];
   // //   ny_dirTSW = &NormalY[dMMP * numberOfBCnodes];
   // //   ny_dirTSE = &NormalY[dPMP * numberOfBCnodes];
   // //   ny_dirTNW = &NormalY[dMPP * numberOfBCnodes];
   // //   ny_dirBNE = &NormalY[dPPM * numberOfBCnodes];
   // //   ny_dirBSW = &NormalY[dMMM * numberOfBCnodes];
   // //   ny_dirBSE = &NormalY[dPMM * numberOfBCnodes];
   // //   ny_dirBNW = &NormalY[dMPM * numberOfBCnodes];
   //   ////////////////////////////////////////////////////////////////////////////////
   // //   real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
   // //           *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
   // //           *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
   // //           *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
   // //           *nz_dirBSE, *nz_dirBNW; 
   // //   nz_dirE   = &NormalZ[dP00 * numberOfBCnodes];
   // //   nz_dirW   = &NormalZ[dM00 * numberOfBCnodes];
   // //   nz_dirN   = &NormalZ[d0P0 * numberOfBCnodes];
   // //   nz_dirS   = &NormalZ[d0M0 * numberOfBCnodes];
   // //   nz_dirT   = &NormalZ[d00P * numberOfBCnodes];
   // //   nz_dirB   = &NormalZ[d00M * numberOfBCnodes];
   // //   nz_dirNE  = &NormalZ[dPP0 * numberOfBCnodes];
   // //   nz_dirSW  = &NormalZ[dMM0 * numberOfBCnodes];
   // //   nz_dirSE  = &NormalZ[dPM0 * numberOfBCnodes];
   // //   nz_dirNW  = &NormalZ[dMP0 * numberOfBCnodes];
   // //   nz_dirTE  = &NormalZ[dP0P * numberOfBCnodes];
   // //   nz_dirBW  = &NormalZ[dM0M * numberOfBCnodes];
   // //   nz_dirBE  = &NormalZ[dP0M * numberOfBCnodes];
   // //   nz_dirTW  = &NormalZ[dM0P * numberOfBCnodes];
   // //   nz_dirTN  = &NormalZ[d0PP * numberOfBCnodes];
   // //   nz_dirBS  = &NormalZ[d0MM * numberOfBCnodes];
   // //   nz_dirBN  = &NormalZ[d0PM * numberOfBCnodes];
   // //   nz_dirTS  = &NormalZ[d0MP * numberOfBCnodes];
   // //   nz_dirTNE = &NormalZ[dPPP * numberOfBCnodes];
   // //   nz_dirTSW = &NormalZ[dMMP * numberOfBCnodes];
   // //   nz_dirTSE = &NormalZ[dPMP * numberOfBCnodes];
   // //   nz_dirTNW = &NormalZ[dMPP * numberOfBCnodes];
   // //   nz_dirBNE = &NormalZ[dPPM * numberOfBCnodes];
   // //   nz_dirBSW = &NormalZ[dMMM * numberOfBCnodes];
   // //   nz_dirBSE = &NormalZ[dPMM * numberOfBCnodes];
   // //   nz_dirBNW = &NormalZ[dMPM * numberOfBCnodes];
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //bool changeCell = false;
   //   unsigned int KQK  = k_Q[k];
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //if( q_dirW[k] > c0o1 || q_dirS[k] > c0o1 || q_dirB[k] > c0o1 ) {
		 // KQK = neighborWSB[KQK];
		 // changeCell = true;
	  //}
	  //if( q_dirW[k] == c0o1 && changeCell == true ) {
		 // KQK = neighborX[KQK];
	  //}
	  //if( q_dirS[k] == c0o1 && changeCell == true ) {
		 // KQK = neighborY[KQK];
	  //}
	  //if( q_dirB[k] == c0o1 && changeCell == true ) {
		 // KQK = neighborZ[KQK];
	  //}

	  ////for(int i = 0; i < numberOfParticles; i++){
		 //// //push back?
	  ////}

   //   ////////////////////////////////////////////////////////////////////////////////
   //   //index
   //   //unsigned int KQK  = k_Q[k];
   //   unsigned int kzero= KQK;
   //   unsigned int ke   = KQK;
   //   unsigned int kw   = neighborX[KQK];
   //   unsigned int kn   = KQK;
   //   unsigned int ks   = neighborY[KQK];
   //   unsigned int kt   = KQK;
   //   unsigned int kb   = neighborZ[KQK];
   //   unsigned int ksw  = neighborY[kw];
   //   unsigned int kne  = KQK;
   //   unsigned int kse  = ks;
   //   unsigned int knw  = kw;
   //   unsigned int kbw  = neighborZ[kw];
   //   unsigned int kte  = KQK;
   //   unsigned int kbe  = kb;
   //   unsigned int ktw  = kw;
   //   unsigned int kbs  = neighborZ[ks];
   //   unsigned int ktn  = KQK;
   //   unsigned int kbn  = kb;
   //   unsigned int kts  = ks;
   //   unsigned int ktse = ks;
   //   unsigned int kbnw = kbw;
   //   unsigned int ktnw = kw;
   //   unsigned int kbse = kbs;
   //   unsigned int ktsw = ksw;
   //   unsigned int kbne = kb;
   //   unsigned int ktne = KQK;
   //   unsigned int kbsw = neighborZ[ksw];
   //   ////////////////////////////////////////////////////////////////////////////////
   //   real f_W    = (D.f[dP00])[ke   ];
   //   real f_E    = (D.f[dM00])[kw   ];
   //   real f_S    = (D.f[d0P0])[kn   ];
   //   real f_N    = (D.f[d0M0])[ks   ];
   //   real f_B    = (D.f[d00P])[kt   ];
   //   real f_T    = (D.f[d00M])[kb   ];
   //   real f_SW   = (D.f[dPP0])[kne  ];
   //   real f_NE   = (D.f[dMM0])[ksw  ];
   //   real f_NW   = (D.f[dPM0])[kse  ];
   //   real f_SE   = (D.f[dMP0])[knw  ];
   //   real f_BW   = (D.f[dP0P])[kte  ];
   //   real f_TE   = (D.f[dM0M])[kbw  ];
   //   real f_TW   = (D.f[dP0M])[kbe  ];
   //   real f_BE   = (D.f[dM0P])[ktw  ];
   //   real f_BS   = (D.f[d0PP])[ktn  ];
   //   real f_TN   = (D.f[d0MM])[kbs  ];
   //   real f_TS   = (D.f[d0PM])[kbn  ];
   //   real f_BN   = (D.f[d0MP])[kts  ];
   //   real f_BSW  = (D.f[dPPP])[ktne ];
   //   real f_BNE  = (D.f[dMMP])[ktsw ];
   //   real f_BNW  = (D.f[dPMP])[ktse ];
   //   real f_BSE  = (D.f[dMPP])[ktnw ];
   //   real f_TSW  = (D.f[dPPM])[kbne ];
   //   real f_TNE  = (D.f[dMMM])[kbsw ];
   //   real f_TNW  = (D.f[dPMM])[kbse ];
   //   real f_TSE  = (D.f[dMPM])[kbnw ];
   //   ////////////////////////////////////////////////////////////////////////////////
   //   // real feq, q;
   //   real vx1, vx2, vx3, drho;
   //   drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
   //             f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
   //             f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]); 

   //   vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
   //             ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
   //             (f_E - f_W)) / (c1o1 + drho); 
   //      

   //   vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
   //              ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
   //              (f_N - f_S)) / (c1o1 + drho); 

   //   vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
   //              (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
   //              (f_T - f_B)) / (c1o1 + drho); 

   //   //real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

   //   //////////////////////////////////////////////////////////////////////////
   //   if (isEvenTimestep==false)
   //   {
   //      D.f[dP00] = &DD[dP00 * size_Mat];
   //      D.f[dM00] = &DD[dM00 * size_Mat];
   //      D.f[d0P0] = &DD[d0P0 * size_Mat];
   //      D.f[d0M0] = &DD[d0M0 * size_Mat];
   //      D.f[d00P] = &DD[d00P * size_Mat];
   //      D.f[d00M] = &DD[d00M * size_Mat];
   //      D.f[dPP0] = &DD[dPP0 * size_Mat];
   //      D.f[dMM0] = &DD[dMM0 * size_Mat];
   //      D.f[dPM0] = &DD[dPM0 * size_Mat];
   //      D.f[dMP0] = &DD[dMP0 * size_Mat];
   //      D.f[dP0P] = &DD[dP0P * size_Mat];
   //      D.f[dM0M] = &DD[dM0M * size_Mat];
   //      D.f[dP0M] = &DD[dP0M * size_Mat];
   //      D.f[dM0P] = &DD[dM0P * size_Mat];
   //      D.f[d0PP] = &DD[d0PP * size_Mat];
   //      D.f[d0MM] = &DD[d0MM * size_Mat];
   //      D.f[d0PM] = &DD[d0PM * size_Mat];
   //      D.f[d0MP] = &DD[d0MP * size_Mat];
   //      D.f[d000] = &DD[d000 * size_Mat];
   //      D.f[dPPP] = &DD[dPPP * size_Mat];
   //      D.f[dMMP] = &DD[dMMP * size_Mat];
   //      D.f[dPMP] = &DD[dPMP * size_Mat];
   //      D.f[dMPP] = &DD[dMPP * size_Mat];
   //      D.f[dPPM] = &DD[dPPM * size_Mat];
   //      D.f[dMMM] = &DD[dMMM * size_Mat];
   //      D.f[dPMM] = &DD[dPMM * size_Mat];
   //      D.f[dMPM] = &DD[dMPM * size_Mat];
   //   } 
   //   else
   //   {
   //      D.f[dM00] = &DD[dP00 * size_Mat];
   //      D.f[dP00] = &DD[dM00 * size_Mat];
   //      D.f[d0M0] = &DD[d0P0 * size_Mat];
   //      D.f[d0P0] = &DD[d0M0 * size_Mat];
   //      D.f[d00M] = &DD[d00P * size_Mat];
   //      D.f[d00P] = &DD[d00M * size_Mat];
   //      D.f[dMM0] = &DD[dPP0 * size_Mat];
   //      D.f[dPP0] = &DD[dMM0 * size_Mat];
   //      D.f[dMP0] = &DD[dPM0 * size_Mat];
   //      D.f[dPM0] = &DD[dMP0 * size_Mat];
   //      D.f[dM0M] = &DD[dP0P * size_Mat];
   //      D.f[dP0P] = &DD[dM0M * size_Mat];
   //      D.f[dM0P] = &DD[dP0M * size_Mat];
   //      D.f[dP0M] = &DD[dM0P * size_Mat];
   //      D.f[d0MM] = &DD[d0PP * size_Mat];
   //      D.f[d0PP] = &DD[d0MM * size_Mat];
   //      D.f[d0MP] = &DD[d0PM * size_Mat];
   //      D.f[d0PM] = &DD[d0MP * size_Mat];
   //      D.f[d000] = &DD[d000 * size_Mat];
   //      D.f[dPPP] = &DD[dMMM * size_Mat];
   //      D.f[dMMP] = &DD[dPPM * size_Mat];
   //      D.f[dPMP] = &DD[dMPM * size_Mat];
   //      D.f[dMPP] = &DD[dPMM * size_Mat];
   //      D.f[dPPM] = &DD[dMMP * size_Mat];
   //      D.f[dMMM] = &DD[dPPP * size_Mat];
   //      D.f[dPMM] = &DD[dMPP * size_Mat];
   //      D.f[dMPM] = &DD[dPMP * size_Mat];
   //   }
   //}
}
