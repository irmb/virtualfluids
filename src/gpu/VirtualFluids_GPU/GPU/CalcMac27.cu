#include "hip/hip_runtime.h"
//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ / 
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;

#include "lbm/MacroscopicQuantities.h"

#include "../Kernel/Utilities/DistributionHelper.cuh"


////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMac27( real* vxD,
                                        real* vyD,
                                        real* vzD,
                                        real* rhoD,
                                        unsigned int* geoD,
                                        unsigned int* neighborX,
                                        unsigned int* neighborY,
                                        unsigned int* neighborZ,
                                        unsigned int size_Mat,
                                        real* distributions,
                                        bool isEvenTimestep)
{
   const unsigned int tx = threadIdx.x;    // Thread index = lokaler i index
   const unsigned int by = blockIdx.x;     // Block index x
   const unsigned int bz = blockIdx.y;     // Block index y
   const unsigned int x = tx + STARTOFFX;  // Globaler x-Index 
   const unsigned int y = by + STARTOFFY;  // Globaler y-Index 
   const unsigned int z = bz + STARTOFFZ;  // Globaler z-Index 

   const unsigned nx = blockDim.x + 2 * STARTOFFX;
   const unsigned ny = gridDim.x + 2 * STARTOFFY;

   const unsigned int k = nx*(ny*z + y) + x; // Zugriff auf arrays im device


   if(k >= size_Mat)
      return;

   if(!vf::gpu::isValidFluidNode(geoD[k]))
      return;

   rhoD[k] = c0o1;
   vxD[k]  = c0o1;
   vyD[k]  = c0o1;
   vzD[k]  = c0o1;

   vf::gpu::DistributionWrapper distr_wrapper(distributions, size_Mat, isEvenTimestep, k, neighborX, neighborY, neighborZ);
   const auto& distribution = distr_wrapper.distribution;

   rhoD[k] = vf::lbm::getDensity(distribution.f);
   vxD[k] = vf::lbm::getIncompressibleVelocityX1(distribution.f);
   vyD[k] = vf::lbm::getIncompressibleVelocityX2(distribution.f);
   vzD[k] = vf::lbm::getIncompressibleVelocityX3(distribution.f);

}





////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacSP27( real* vxD,
                                          real* vyD,
                                          real* vzD,
                                          real* rhoD,
                                          real* pressD,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD,
                                          bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
	  rhoD[k]   = c0o1;
	  vxD[k]    = c0o1;
	  vyD[k]    = c0o1;
	  vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vxD[k]     =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         vyD[k]     =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vzD[k]     =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         pressD[k]  =  ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        2.f*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        3.f*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1+c0o1*rhoD[k])) * c1o2+rhoD[k]; // times zero for incompressible case   
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5

      }
   }
}


////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacCompSP27(real *vxD, real *vyD, real *vzD, real *rhoD, real *pressD,
                                             unsigned int *geoD, unsigned int *neighborX, unsigned int *neighborY,
                                             unsigned int *neighborZ, unsigned int size_Mat, real *distributions,
                                             bool isEvenTimestep)
{
    const unsigned k = vf::gpu::getNodeIndex();

    if(k >= size_Mat)
        return;

    pressD[k] = c0o1;
    rhoD[k]   = c0o1;
    vxD[k]    = c0o1;
    vyD[k]    = c0o1;
    vzD[k]    = c0o1;

    if (!vf::gpu::isValidFluidNode(geoD[k]))
        return;

    vf::gpu::DistributionWrapper distr_wrapper(distributions, size_Mat, isEvenTimestep, k, neighborX, neighborY,
                                               neighborZ);
    const auto &distribution = distr_wrapper.distribution;

    rhoD[k]   = vf::lbm::getDensity(distribution.f);
    vxD[k]    = vf::lbm::getCompressibleVelocityX1(distribution.f, rhoD[k]);
    vyD[k]    = vf::lbm::getCompressibleVelocityX2(distribution.f, rhoD[k]);
    vzD[k]    = vf::lbm::getCompressibleVelocityX3(distribution.f, rhoD[k]);
    pressD[k] = vf::lbm::getPressure(distribution.f, rhoD[k], vxD[k], vyD[k], vzD[k]); 
}




































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMedSP27( real* vxD,
                                          real* vyD,
                                          real* vzD,
                                          real* rhoD,
                                          real* pressD,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD,
                                          bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
	  rhoD[k]   = c0o1;
	  vxD[k]    = c0o1;
	  vyD[k]    = c0o1;
	  vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]+
                        RHO;

         vxD[k]     =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]+
                        VX;

         vyD[k]     =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]+
                        VY;

         vzD[k]     =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]+
                        VZ;

         pressD[k]  =   ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        c2o1*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        c3o1*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1+rhoD[k])) * c1o2+rhoD[k]+
                        PRESS;    
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMedCompSP27( real* vxD,
											  real* vyD,
											  real* vzD,
											  real* rhoD,
											  real* pressD,
											  unsigned int* geoD,
											  unsigned int* neighborX,
											  unsigned int* neighborY,
											  unsigned int* neighborZ,
											  unsigned int size_Mat,
											  real* DD,
											  bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      //unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
	  rhoD[k]   = c0o1;
	  vxD[k]    = c0o1;
	  vyD[k]    = c0o1;
	  vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
		  real mfcbb = (D.f[dirE])[k];//[ke   ];
		  real mfabb = (D.f[dirW])[kw];//[kw   ];  
		  real mfbcb = (D.f[dirN])[k];//[kn   ];
		  real mfbab = (D.f[dirS])[ks];//[ks   ];  
		  real mfbbc = (D.f[dirT])[k];//[kt   ];
		  real mfbba = (D.f[dirB])[kb];//[kb   ];  
		  real mfccb = (D.f[dirNE])[k];//[kne  ];  
		  real mfaab = (D.f[dirSW])[ksw];//[ksw  ];
		  real mfcab = (D.f[dirSE])[ks];//[kse  ]; 
		  real mfacb = (D.f[dirNW])[kw];//[knw  ]; 
		  real mfcbc = (D.f[dirTE])[k];//[kte  ];  
		  real mfaba = (D.f[dirBW])[kbw];//[kbw  ];
		  real mfcba = (D.f[dirBE])[kb];//[kbe  ]; 
		  real mfabc = (D.f[dirTW])[kw];//[ktw  ]; 
		  real mfbcc = (D.f[dirTN])[k];//[ktn  ];  
		  real mfbaa = (D.f[dirBS])[kbs];//[kbs  ];
		  real mfbca = (D.f[dirBN])[kb];//[kbn  ]; 
		  real mfbac = (D.f[dirTS])[ks];//[kts  ]; 
		  real mfbbb = (D.f[dirZERO])[k];//[kzero];
		  real mfccc = (D.f[dirTNE])[k];//[ktne ]; 
		  real mfaac = (D.f[dirTSW])[ksw];//[ktsw ]; 
		  real mfcac = (D.f[dirTSE])[ks];//[ktse ];
		  real mfacc = (D.f[dirTNW])[kw];//[ktnw ];
		  real mfcca = (D.f[dirBNE])[kb];//[kbne ];
		  real mfaaa = (D.f[dirBSW])[kbsw];//[kbsw ];
		  real mfcaa = (D.f[dirBSE])[kbs];//[kbse ]; 
		  real mfaca = (D.f[dirBNW])[kbw];//[kbnw ]; 
		  ////////////////////////////////////////////////////////////////////////////////////
		  real drho = 
			  ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
			  (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
			  ((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;

		  real rho = c1o1 + drho;
		  
		  rhoD[k] = drho + RHO;

		  vxD[k] = 
			  (((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
			  (((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
			  (mfcbb - mfabb)) / rho) + VX;
		  vyD[k] = 
			  (((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
			  (((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
			  (mfbcb - mfbab)) / rho) + VY;
		  vzD[k] = 
			  (((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
			  (((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
			  (mfbbc - mfbba)) / rho) + VZ;

		  //rhoD[k] =
			 // (D.f[dirE])[ke] + (D.f[dirW])[kw] +
			 // (D.f[dirN])[kn] + (D.f[dirS])[ks] +
			 // (D.f[dirT])[kt] + (D.f[dirB])[kb] +
			 // (D.f[dirNE])[kne] + (D.f[dirSW])[ksw] +
			 // (D.f[dirSE])[kse] + (D.f[dirNW])[knw] +
			 // (D.f[dirTE])[kte] + (D.f[dirBW])[kbw] +
			 // (D.f[dirBE])[kbe] + (D.f[dirTW])[ktw] +
			 // (D.f[dirTN])[ktn] + (D.f[dirBS])[kbs] +
			 // (D.f[dirBN])[kbn] + (D.f[dirTS])[kts] +
			 // (D.f[dirZERO])[kzero] +
			 // (D.f[dirTNE])[ktne] + (D.f[dirTSW])[ktsw] +
			 // (D.f[dirTSE])[ktse] + (D.f[dirTNW])[ktnw] +
			 // (D.f[dirBNE])[kbne] + (D.f[dirBSW])[kbsw] +
			 // (D.f[dirBSE])[kbse] + (D.f[dirBNW])[kbnw];// +RHO;

    //     vxD[k] =  
			 //((D.f[dirE  ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
    //         (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
    //         (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
    //         (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
    //         (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
    //         (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
    //         (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
    //         (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
    //         (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
    //         VX;

    //     vyD[k] =  
			 //((D.f[dirN  ])[kn  ]- (D.f[dirS   ])[ks  ]+
    //         (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
    //         (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
    //         (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
    //         (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
    //         (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
    //         (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
    //         (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
    //         (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
    //         VY;

    //     vzD[k] =  
			 //((D.f[dirT  ])[kt  ]- (D.f[dirB   ])[kb  ]+
    //         (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
    //         (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
    //         (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
    //         (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
    //         (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
    //         (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
    //         (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
    //         (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
    //         VZ;

         pressD[k]  =  ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        c2o1*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        c3o1*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1+rhoD[k])) * c1o2+rhoD[k]+
                        PRESS;    
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMedCompAD27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	real* concD,
	unsigned int* geoD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	real* DD,
	real* DD_AD,
	bool isEvenTimestep)
{
	Distributions27 D;
	if (isEvenTimestep == true)
	{
		D.f[dirE] = &DD[dirE   *size_Mat];
		D.f[dirW] = &DD[dirW   *size_Mat];
		D.f[dirN] = &DD[dirN   *size_Mat];
		D.f[dirS] = &DD[dirS   *size_Mat];
		D.f[dirT] = &DD[dirT   *size_Mat];
		D.f[dirB] = &DD[dirB   *size_Mat];
		D.f[dirNE] = &DD[dirNE  *size_Mat];
		D.f[dirSW] = &DD[dirSW  *size_Mat];
		D.f[dirSE] = &DD[dirSE  *size_Mat];
		D.f[dirNW] = &DD[dirNW  *size_Mat];
		D.f[dirTE] = &DD[dirTE  *size_Mat];
		D.f[dirBW] = &DD[dirBW  *size_Mat];
		D.f[dirBE] = &DD[dirBE  *size_Mat];
		D.f[dirTW] = &DD[dirTW  *size_Mat];
		D.f[dirTN] = &DD[dirTN  *size_Mat];
		D.f[dirBS] = &DD[dirBS  *size_Mat];
		D.f[dirBN] = &DD[dirBN  *size_Mat];
		D.f[dirTS] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE] = &DD[dirTNE *size_Mat];
		D.f[dirTSW] = &DD[dirTSW *size_Mat];
		D.f[dirTSE] = &DD[dirTSE *size_Mat];
		D.f[dirTNW] = &DD[dirTNW *size_Mat];
		D.f[dirBNE] = &DD[dirBNE *size_Mat];
		D.f[dirBSW] = &DD[dirBSW *size_Mat];
		D.f[dirBSE] = &DD[dirBSE *size_Mat];
		D.f[dirBNW] = &DD[dirBNW *size_Mat];
	}
	else
	{
		D.f[dirW] = &DD[dirE   *size_Mat];
		D.f[dirE] = &DD[dirW   *size_Mat];
		D.f[dirS] = &DD[dirN   *size_Mat];
		D.f[dirN] = &DD[dirS   *size_Mat];
		D.f[dirB] = &DD[dirT   *size_Mat];
		D.f[dirT] = &DD[dirB   *size_Mat];
		D.f[dirSW] = &DD[dirNE  *size_Mat];
		D.f[dirNE] = &DD[dirSW  *size_Mat];
		D.f[dirNW] = &DD[dirSE  *size_Mat];
		D.f[dirSE] = &DD[dirNW  *size_Mat];
		D.f[dirBW] = &DD[dirTE  *size_Mat];
		D.f[dirTE] = &DD[dirBW  *size_Mat];
		D.f[dirTW] = &DD[dirBE  *size_Mat];
		D.f[dirBE] = &DD[dirTW  *size_Mat];
		D.f[dirBS] = &DD[dirTN  *size_Mat];
		D.f[dirTN] = &DD[dirBS  *size_Mat];
		D.f[dirTS] = &DD[dirBN  *size_Mat];
		D.f[dirBN] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE] = &DD[dirBSW *size_Mat];
		D.f[dirTSW] = &DD[dirBNE *size_Mat];
		D.f[dirTSE] = &DD[dirBNW *size_Mat];
		D.f[dirTNW] = &DD[dirBSE *size_Mat];
		D.f[dirBNE] = &DD[dirTSW *size_Mat];
		D.f[dirBSW] = &DD[dirTNE *size_Mat];
		D.f[dirBSE] = &DD[dirTNW *size_Mat];
		D.f[dirBNW] = &DD[dirTSE *size_Mat];
	}
	////////////////////////////////////////////////////////////////////////////////
	Distributions27 Dad;
	if (isEvenTimestep == true)
	{
		Dad.f[dirE]    = &DD_AD[dirE   *size_Mat];
		Dad.f[dirW]    = &DD_AD[dirW   *size_Mat];
		Dad.f[dirN]    = &DD_AD[dirN   *size_Mat];
		Dad.f[dirS]    = &DD_AD[dirS   *size_Mat];
		Dad.f[dirT]    = &DD_AD[dirT   *size_Mat];
		Dad.f[dirB]    = &DD_AD[dirB   *size_Mat];
		Dad.f[dirNE]   = &DD_AD[dirNE  *size_Mat];
		Dad.f[dirSW]   = &DD_AD[dirSW  *size_Mat];
		Dad.f[dirSE]   = &DD_AD[dirSE  *size_Mat];
		Dad.f[dirNW]   = &DD_AD[dirNW  *size_Mat];
		Dad.f[dirTE]   = &DD_AD[dirTE  *size_Mat];
		Dad.f[dirBW]   = &DD_AD[dirBW  *size_Mat];
		Dad.f[dirBE]   = &DD_AD[dirBE  *size_Mat];
		Dad.f[dirTW]   = &DD_AD[dirTW  *size_Mat];
		Dad.f[dirTN]   = &DD_AD[dirTN  *size_Mat];
		Dad.f[dirBS]   = &DD_AD[dirBS  *size_Mat];
		Dad.f[dirBN]   = &DD_AD[dirBN  *size_Mat];
		Dad.f[dirTS]   = &DD_AD[dirTS  *size_Mat];
		Dad.f[dirZERO] = &DD_AD[dirZERO*size_Mat];
		Dad.f[dirTNE]  = &DD_AD[dirTNE *size_Mat];
		Dad.f[dirTSW]  = &DD_AD[dirTSW *size_Mat];
		Dad.f[dirTSE]  = &DD_AD[dirTSE *size_Mat];
		Dad.f[dirTNW]  = &DD_AD[dirTNW *size_Mat];
		Dad.f[dirBNE]  = &DD_AD[dirBNE *size_Mat];
		Dad.f[dirBSW]  = &DD_AD[dirBSW *size_Mat];
		Dad.f[dirBSE]  = &DD_AD[dirBSE *size_Mat];
		Dad.f[dirBNW]  = &DD_AD[dirBNW *size_Mat];
	}						
	else					
	{						
		Dad.f[dirW]    = &DD_AD[dirE   *size_Mat];
		Dad.f[dirE]    = &DD_AD[dirW   *size_Mat];
		Dad.f[dirS]    = &DD_AD[dirN   *size_Mat];
		Dad.f[dirN]    = &DD_AD[dirS   *size_Mat];
		Dad.f[dirB]    = &DD_AD[dirT   *size_Mat];
		Dad.f[dirT]    = &DD_AD[dirB   *size_Mat];
		Dad.f[dirSW]   = &DD_AD[dirNE  *size_Mat];
		Dad.f[dirNE]   = &DD_AD[dirSW  *size_Mat];
		Dad.f[dirNW]   = &DD_AD[dirSE  *size_Mat];
		Dad.f[dirSE]   = &DD_AD[dirNW  *size_Mat];
		Dad.f[dirBW]   = &DD_AD[dirTE  *size_Mat];
		Dad.f[dirTE]   = &DD_AD[dirBW  *size_Mat];
		Dad.f[dirTW]   = &DD_AD[dirBE  *size_Mat];
		Dad.f[dirBE]   = &DD_AD[dirTW  *size_Mat];
		Dad.f[dirBS]   = &DD_AD[dirTN  *size_Mat];
		Dad.f[dirTN]   = &DD_AD[dirBS  *size_Mat];
		Dad.f[dirTS]   = &DD_AD[dirBN  *size_Mat];
		Dad.f[dirBN]   = &DD_AD[dirTS  *size_Mat];
		Dad.f[dirZERO] = &DD_AD[dirZERO*size_Mat];
		Dad.f[dirTNE]  = &DD_AD[dirBSW *size_Mat];
		Dad.f[dirTSW]  = &DD_AD[dirBNE *size_Mat];
		Dad.f[dirTSE]  = &DD_AD[dirBNW *size_Mat];
		Dad.f[dirTNW]  = &DD_AD[dirBSE *size_Mat];
		Dad.f[dirBNE]  = &DD_AD[dirTSW *size_Mat];
		Dad.f[dirBSW]  = &DD_AD[dirTNE *size_Mat];
		Dad.f[dirBSE]  = &DD_AD[dirTNW *size_Mat];
		Dad.f[dirBNW]  = &DD_AD[dirTSE *size_Mat];
	}
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k < size_Mat)
	{
		//////////////////////////////////////////////////////////////////////////
		//index
		//unsigned int kzero = k;
		unsigned int ke = k;
		unsigned int kw = neighborX[k];
		unsigned int kn = k;
		unsigned int ks = neighborY[k];
		unsigned int kt = k;
		unsigned int kb = neighborZ[k];
		unsigned int ksw = neighborY[kw];
		unsigned int kne = k;
		unsigned int kse = ks;
		unsigned int knw = kw;
		unsigned int kbw = neighborZ[kw];
		unsigned int kte = k;
		unsigned int kbe = kb;
		unsigned int ktw = kw;
		unsigned int kbs = neighborZ[ks];
		unsigned int ktn = k;
		unsigned int kbn = kb;
		unsigned int kts = ks;
		unsigned int ktse = ks;
		unsigned int kbnw = kbw;
		unsigned int ktnw = kw;
		unsigned int kbse = kbs;
		unsigned int ktsw = ksw;
		unsigned int kbne = kb;
		unsigned int ktne = k;
		unsigned int kbsw = neighborZ[ksw];
		//////////////////////////////////////////////////////////////////////////
		real CONC  = concD[k];
		real PRESS = pressD[k];
		real RHO   = rhoD[k];
		real VX    = vxD[k];
		real VY    = vyD[k];
		real VZ    = vzD[k];
		//////////////////////////////////////////////////////////////////////////
		concD[k] = c0o1;
		pressD[k] = c0o1;
		rhoD[k] = c0o1;
		vxD[k] = c0o1;
		vyD[k] = c0o1;
		vzD[k] = c0o1;

		if (geoD[k] == GEO_FLUID)
		{
			real mfcbb = (D.f[dirE])[k];//[ke   ];
			real mfabb = (D.f[dirW])[kw];//[kw   ];  
			real mfbcb = (D.f[dirN])[k];//[kn   ];
			real mfbab = (D.f[dirS])[ks];//[ks   ];  
			real mfbbc = (D.f[dirT])[k];//[kt   ];
			real mfbba = (D.f[dirB])[kb];//[kb   ];  
			real mfccb = (D.f[dirNE])[k];//[kne  ];  
			real mfaab = (D.f[dirSW])[ksw];//[ksw  ];
			real mfcab = (D.f[dirSE])[ks];//[kse  ]; 
			real mfacb = (D.f[dirNW])[kw];//[knw  ]; 
			real mfcbc = (D.f[dirTE])[k];//[kte  ];  
			real mfaba = (D.f[dirBW])[kbw];//[kbw  ];
			real mfcba = (D.f[dirBE])[kb];//[kbe  ]; 
			real mfabc = (D.f[dirTW])[kw];//[ktw  ]; 
			real mfbcc = (D.f[dirTN])[k];//[ktn  ];  
			real mfbaa = (D.f[dirBS])[kbs];//[kbs  ];
			real mfbca = (D.f[dirBN])[kb];//[kbn  ]; 
			real mfbac = (D.f[dirTS])[ks];//[kts  ]; 
			real mfbbb = (D.f[dirZERO])[k];//[kzero];
			real mfccc = (D.f[dirTNE])[k];//[ktne ]; 
			real mfaac = (D.f[dirTSW])[ksw];//[ktsw ]; 
			real mfcac = (D.f[dirTSE])[ks];//[ktse ];
			real mfacc = (D.f[dirTNW])[kw];//[ktnw ];
			real mfcca = (D.f[dirBNE])[kb];//[kbne ];
			real mfaaa = (D.f[dirBSW])[kbsw];//[kbsw ];
			real mfcaa = (D.f[dirBSE])[kbs];//[kbse ]; 
			real mfaca = (D.f[dirBNW])[kbw];//[kbnw ]; 
			////////////////////////////////////////////////////////////////////////////////////
			real drho =
				((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				 (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				  ((mfabb + mfcbb) + (mfbab + mfbcb)  +  (mfbba + mfbbc))) + mfbbb;
			real rho = c1o1 + drho;
			////////////////////////////////////////////////////////////////////////////////////

			rhoD[k] = drho + RHO;

			vxD[k] =
				(((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
				(((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
					(mfcbb - mfabb)) / rho) + VX;
			
			vyD[k] =
				(((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
				(((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
					(mfbcb - mfbab)) / rho) + VY;
			
			vzD[k] =
				(((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
				(((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
					(mfbbc - mfbba)) / rho) + VZ;

			pressD[k] = 
				((D.f[dirE])[ke] + (D.f[dirW])[kw] +
				 (D.f[dirN])[kn] + (D.f[dirS])[ks] +
				 (D.f[dirT])[kt] + (D.f[dirB])[kb] +
				 c2o1*(
				 (D.f[dirNE])[kne] + (D.f[dirSW])[ksw] +
				 (D.f[dirSE])[kse] + (D.f[dirNW])[knw] +
				 (D.f[dirTE])[kte] + (D.f[dirBW])[kbw] +
				 (D.f[dirBE])[kbe] + (D.f[dirTW])[ktw] +
				 (D.f[dirTN])[ktn] + (D.f[dirBS])[kbs] +
				 (D.f[dirBN])[kbn] + (D.f[dirTS])[kts]) +
				 c3o1*(
				 (D.f[dirTNE])[ktne] + (D.f[dirTSW])[ktsw] +
				 (D.f[dirTSE])[ktse] + (D.f[dirTNW])[ktnw] +
				 (D.f[dirBNE])[kbne] + (D.f[dirBSW])[kbsw] +
				 (D.f[dirBSE])[kbse] + (D.f[dirBNW])[kbnw]) -
				 rhoD[k] - (vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1 + rhoD[k])) * c1o2 + rhoD[k] +
				 PRESS;
				 //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5
			//////////////////////////////////////////////////////////////////////////
			mfcbb = (Dad.f[dirE   ])[k   ];
			mfabb = (Dad.f[dirW   ])[kw  ];
			mfbcb = (Dad.f[dirN   ])[k   ];
			mfbab = (Dad.f[dirS   ])[ks  ];
			mfbbc = (Dad.f[dirT   ])[k   ];
			mfbba = (Dad.f[dirB   ])[kb  ];
			mfccb = (Dad.f[dirNE  ])[k   ];
			mfaab = (Dad.f[dirSW  ])[ksw ];
			mfcab = (Dad.f[dirSE  ])[ks  ];
			mfacb = (Dad.f[dirNW  ])[kw  ];
			mfcbc = (Dad.f[dirTE  ])[k   ];
			mfaba = (Dad.f[dirBW  ])[kbw ];
			mfcba = (Dad.f[dirBE  ])[kb  ];
			mfabc = (Dad.f[dirTW  ])[kw  ];
			mfbcc = (Dad.f[dirTN  ])[k   ];
			mfbaa = (Dad.f[dirBS  ])[kbs ];
			mfbca = (Dad.f[dirBN  ])[kb  ];
			mfbac = (Dad.f[dirTS  ])[ks  ];
			mfbbb = (Dad.f[dirZERO])[k   ];
			mfccc = (Dad.f[dirTNE ])[k   ];
			mfaac = (Dad.f[dirTSW ])[ksw ];
			mfcac = (Dad.f[dirTSE ])[ks  ];
			mfacc = (Dad.f[dirTNW ])[kw  ];
			mfcca = (Dad.f[dirBNE ])[kb  ];
			mfaaa = (Dad.f[dirBSW ])[kbsw];
			mfcaa = (Dad.f[dirBSE ])[kbs ];
			mfaca = (Dad.f[dirBNW ])[kbw ];
			//////////////////////////////////////////////////////////////////////////
			concD[k] = 
				((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa)   + (mfaac + mfcca))) +
				 (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba)   + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				  ((mfabb + mfcbb) + (mfbab + mfbcb)  +  (mfbba + mfbbc))) +  mfbbb + CONC;
		}
	}
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacMedSP27( real* vxD,
                                             real* vyD,
                                             real* vzD,
                                             real* rhoD,
                                             real* pressD,
                                             unsigned int* geoD,
                                             unsigned int* neighborX,
                                             unsigned int* neighborY,
                                             unsigned int* neighborZ,
                                             unsigned int tdiff,
                                             unsigned int size_Mat,
                                             bool isEvenTimestep)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
      rhoD[k]   = c0o1;
      vxD[k]    = c0o1;
      vyD[k]    = c0o1;
      vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   RHO   / tdiff;
         vxD[k]     =   VX    / tdiff;
         vyD[k]     =   VY    / tdiff;
         vzD[k]     =   VZ    / tdiff;
         pressD[k]  =   PRESS / tdiff;    
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBResetMedianValuesSP27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	unsigned int size_Mat,
	bool isEvenTimestep)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k<size_Mat)
	{
		//////////////////////////////////////////////////////////////////////////
		pressD[k] = c0o1;
		rhoD[k] = c0o1;
		vxD[k] = c0o1;
		vyD[k] = c0o1;
		vzD[k] = c0o1;
	}
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBResetMedianValuesAD27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	real* concD,
	unsigned int size_Mat,
	bool isEvenTimestep)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k < size_Mat)
	{
		//////////////////////////////////////////////////////////////////////////
		concD[k]  = c0o1;
		pressD[k] = c0o1;
		rhoD[k]   = c0o1;
		vxD[k]    = c0o1;
		vyD[k]    = c0o1;
		vzD[k]    = c0o1;
	}
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMeasurePoints( real* vxMP,
												real* vyMP,
												real* vzMP,
												real* rhoMP,
												unsigned int* kMP,
												unsigned int numberOfPointskMP,
												unsigned int MPClockCycle,
												unsigned int t,
												unsigned int* geoD,
												unsigned int* neighborX,
												unsigned int* neighborY,
												unsigned int* neighborZ,
												unsigned int size_Mat,
												real* DD,
												bool isEvenTimestep)
{
	Distributions27 D;
	if (isEvenTimestep==true)
	{
		D.f[dirE   ] = &DD[dirE   *size_Mat];
		D.f[dirW   ] = &DD[dirW   *size_Mat];
		D.f[dirN   ] = &DD[dirN   *size_Mat];
		D.f[dirS   ] = &DD[dirS   *size_Mat];
		D.f[dirT   ] = &DD[dirT   *size_Mat];
		D.f[dirB   ] = &DD[dirB   *size_Mat];
		D.f[dirNE  ] = &DD[dirNE  *size_Mat];
		D.f[dirSW  ] = &DD[dirSW  *size_Mat];
		D.f[dirSE  ] = &DD[dirSE  *size_Mat];
		D.f[dirNW  ] = &DD[dirNW  *size_Mat];
		D.f[dirTE  ] = &DD[dirTE  *size_Mat];
		D.f[dirBW  ] = &DD[dirBW  *size_Mat];
		D.f[dirBE  ] = &DD[dirBE  *size_Mat];
		D.f[dirTW  ] = &DD[dirTW  *size_Mat];
		D.f[dirTN  ] = &DD[dirTN  *size_Mat];
		D.f[dirBS  ] = &DD[dirBS  *size_Mat];
		D.f[dirBN  ] = &DD[dirBN  *size_Mat];
		D.f[dirTS  ] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE ] = &DD[dirTNE *size_Mat];
		D.f[dirTSW ] = &DD[dirTSW *size_Mat];
		D.f[dirTSE ] = &DD[dirTSE *size_Mat];
		D.f[dirTNW ] = &DD[dirTNW *size_Mat];
		D.f[dirBNE ] = &DD[dirBNE *size_Mat];
		D.f[dirBSW ] = &DD[dirBSW *size_Mat];
		D.f[dirBSE ] = &DD[dirBSE *size_Mat];
		D.f[dirBNW ] = &DD[dirBNW *size_Mat];
	} 
	else
	{
		D.f[dirW   ] = &DD[dirE   *size_Mat];
		D.f[dirE   ] = &DD[dirW   *size_Mat];
		D.f[dirS   ] = &DD[dirN   *size_Mat];
		D.f[dirN   ] = &DD[dirS   *size_Mat];
		D.f[dirB   ] = &DD[dirT   *size_Mat];
		D.f[dirT   ] = &DD[dirB   *size_Mat];
		D.f[dirSW  ] = &DD[dirNE  *size_Mat];
		D.f[dirNE  ] = &DD[dirSW  *size_Mat];
		D.f[dirNW  ] = &DD[dirSE  *size_Mat];
		D.f[dirSE  ] = &DD[dirNW  *size_Mat];
		D.f[dirBW  ] = &DD[dirTE  *size_Mat];
		D.f[dirTE  ] = &DD[dirBW  *size_Mat];
		D.f[dirTW  ] = &DD[dirBE  *size_Mat];
		D.f[dirBE  ] = &DD[dirTW  *size_Mat];
		D.f[dirBS  ] = &DD[dirTN  *size_Mat];
		D.f[dirTN  ] = &DD[dirBS  *size_Mat];
		D.f[dirTS  ] = &DD[dirBN  *size_Mat];
		D.f[dirBN  ] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE ] = &DD[dirBSW *size_Mat];
		D.f[dirTSW ] = &DD[dirBNE *size_Mat];
		D.f[dirTSE ] = &DD[dirBNW *size_Mat];
		D.f[dirTNW ] = &DD[dirBSE *size_Mat];
		D.f[dirBNE ] = &DD[dirTSW *size_Mat];
		D.f[dirBSW ] = &DD[dirTNE *size_Mat];
		D.f[dirBSE ] = &DD[dirTNW *size_Mat];
		D.f[dirBNW ] = &DD[dirTSE *size_Mat];
	}
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if(k<numberOfPointskMP)
	{
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kMP[k];//k;
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = kzero;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = kzero;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = kzero;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = kzero;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
	  unsigned int kMac = k*MPClockCycle + t;
	  //////////////////////////////////////////////////////////////////////////

      if(geoD[kzero] == GEO_FLUID)
      {
         rhoMP[kMac]=   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vxMP[kMac] =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         vyMP[kMac] =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vzMP[kMac] =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBSetOutputWallVelocitySP27( real* vxD,
														real* vyD,
														real* vzD,
														real* vxWall,
														real* vyWall,
														real* vzWall,
														int numberOfWallNodes, 
														int* kWallNodes, 
														real* rhoD,
														real* pressD,
														unsigned int* geoD,
														unsigned int* neighborX,
														unsigned int* neighborY,
														unsigned int* neighborZ,
														unsigned int size_Mat,
														real* DD,
														bool isEvenTimestep)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfWallNodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KWN  = kWallNodes[k];
      //////////////////////////////////////////////////////////////////////////
      vxD[KWN] = 0.0;//vxWall[k];
      vyD[KWN] = 0.0;//vyWall[k];
      vzD[KWN] = 0.0;//vzWall[k];
   }
}





























