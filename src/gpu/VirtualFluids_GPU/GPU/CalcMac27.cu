#include "hip/hip_runtime.h"
//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ / 
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include "Core/RealConstants.h"

#include "lbm/CalcMac.h"


__device__ Distributions27 getDistributions(real* DD, unsigned int size_Mat, bool evenOrOdd)
{
    Distributions27 D;
    if (evenOrOdd)
    {
        D.f[dirE   ] = &DD[dirE   *size_Mat];
        D.f[dirW   ] = &DD[dirW   *size_Mat];
        D.f[dirN   ] = &DD[dirN   *size_Mat];
        D.f[dirS   ] = &DD[dirS   *size_Mat];
        D.f[dirT   ] = &DD[dirT   *size_Mat];
        D.f[dirB   ] = &DD[dirB   *size_Mat];
        D.f[dirNE  ] = &DD[dirNE  *size_Mat];
        D.f[dirSW  ] = &DD[dirSW  *size_Mat];
        D.f[dirSE  ] = &DD[dirSE  *size_Mat];
        D.f[dirNW  ] = &DD[dirNW  *size_Mat];
        D.f[dirTE  ] = &DD[dirTE  *size_Mat];
        D.f[dirBW  ] = &DD[dirBW  *size_Mat];
        D.f[dirBE  ] = &DD[dirBE  *size_Mat];
        D.f[dirTW  ] = &DD[dirTW  *size_Mat];
        D.f[dirTN  ] = &DD[dirTN  *size_Mat];
        D.f[dirBS  ] = &DD[dirBS  *size_Mat];
        D.f[dirBN  ] = &DD[dirBN  *size_Mat];
        D.f[dirTS  ] = &DD[dirTS  *size_Mat];
        D.f[dirZERO] = &DD[dirZERO*size_Mat];
        D.f[dirTNE ] = &DD[dirTNE *size_Mat];
        D.f[dirTSW ] = &DD[dirTSW *size_Mat];
        D.f[dirTSE ] = &DD[dirTSE *size_Mat];
        D.f[dirTNW ] = &DD[dirTNW *size_Mat];
        D.f[dirBNE ] = &DD[dirBNE *size_Mat];
        D.f[dirBSW ] = &DD[dirBSW *size_Mat];
        D.f[dirBSE ] = &DD[dirBSE *size_Mat];
        D.f[dirBNW ] = &DD[dirBNW *size_Mat];
    }
    else
    {
        D.f[dirW   ] = &DD[dirE   *size_Mat];
        D.f[dirE   ] = &DD[dirW   *size_Mat];
        D.f[dirS   ] = &DD[dirN   *size_Mat];
        D.f[dirN   ] = &DD[dirS   *size_Mat];
        D.f[dirB   ] = &DD[dirT   *size_Mat];
        D.f[dirT   ] = &DD[dirB   *size_Mat];
        D.f[dirSW  ] = &DD[dirNE  *size_Mat];
        D.f[dirNE  ] = &DD[dirSW  *size_Mat];
        D.f[dirNW  ] = &DD[dirSE  *size_Mat];
        D.f[dirSE  ] = &DD[dirNW  *size_Mat];
        D.f[dirBW  ] = &DD[dirTE  *size_Mat];
        D.f[dirTE  ] = &DD[dirBW  *size_Mat];
        D.f[dirTW  ] = &DD[dirBE  *size_Mat];
        D.f[dirBE  ] = &DD[dirTW  *size_Mat];
        D.f[dirBS  ] = &DD[dirTN  *size_Mat];
        D.f[dirTN  ] = &DD[dirBS  *size_Mat];
        D.f[dirTS  ] = &DD[dirBN  *size_Mat];
        D.f[dirBN  ] = &DD[dirTS  *size_Mat];
        D.f[dirZERO] = &DD[dirZERO*size_Mat];
        D.f[dirTNE ] = &DD[dirBSW *size_Mat];
        D.f[dirTSW ] = &DD[dirBNE *size_Mat];
        D.f[dirTSE ] = &DD[dirBNW *size_Mat];
        D.f[dirTNW ] = &DD[dirBSE *size_Mat];
        D.f[dirBNE ] = &DD[dirTSW *size_Mat];
        D.f[dirBSW ] = &DD[dirTNE *size_Mat];
        D.f[dirBSE ] = &DD[dirTNW *size_Mat];
        D.f[dirBNW ] = &DD[dirTSE *size_Mat];
    }
    return D;
}

struct Distribution27
{
   real f[27];
};

__device__ Distribution27 getDistribution(real* DD, unsigned int size_Mat, bool evenOrOdd, unsigned int k,
                                    unsigned int* neighborX,
                                    unsigned int* neighborY,
                                    unsigned int* neighborZ)
{
    unsigned int kzero= k;
    unsigned int ke   = k;
    unsigned int kw   = neighborX[k];
    unsigned int kn   = k;
    unsigned int ks   = neighborY[k];
    unsigned int kt   = k;
    unsigned int kb   = neighborZ[k];
    unsigned int ksw  = neighborY[kw];
    unsigned int kne  = k;
    unsigned int kse  = ks;
    unsigned int knw  = kw;
    unsigned int kbw  = neighborZ[kw];
    unsigned int kte  = k;
    unsigned int kbe  = kb;
    unsigned int ktw  = kw;
    unsigned int kbs  = neighborZ[ks];
    unsigned int ktn  = k;
    unsigned int kbn  = kb;
    unsigned int kts  = ks;
    unsigned int ktse = ks;
    unsigned int kbnw = kbw;
    unsigned int ktnw = kw;
    unsigned int kbse = kbs;
    unsigned int ktsw = ksw;
    unsigned int kbne = kb;
    unsigned int ktne = k;
    unsigned int kbsw = neighborZ[ksw];

    Distributions27 D = getDistributions(DD, size_Mat, evenOrOdd);

//       real f_dirE = (D.f[dirE])[ke];
//       real f_dirW = (D.f[dirW])[kw];
//       real f_dirN = (D.f[dirN])[kn];
//       real f_dirS = (D.f[dirS])[ks];
//       real f_dirT = (D.f[dirT])[kt];
//       real f_dirB = (D.f[dirB])[kb];
//
//       real f_dirNE = (D.f[dirNE])[kne];
//       real f_dirSW = (D.f[dirSW])[ksw];
//       real f_dirSE = (D.f[dirSE])[kse];
//       real f_dirNW = (D.f[dirNW])[knw];
//       real f_dirTE = (D.f[dirTE])[kte];
//       real f_dirBW = (D.f[dirBW])[kbw];
//
//       real f_dirBE = (D.f[dirBE])[kbe];
//       real f_dirTW = (D.f[dirTW])[ktw];
//       real f_dirTN = (D.f[dirTN])[ktn];
//       real f_dirBS = (D.f[dirBS])[kbs];
//       real f_dirBN = (D.f[dirBN])[kbn];
//       real f_dirTS = (D.f[dirTS])[kts];
//
//       real f_dirZERO = (D.f[dirZERO])[kzero];
//
//       real f_dirTNE = (D.f[dirTNE ])[ktne];
//       real f_dirTSW = (D.f[dirTSW ])[ktsw];
//       real f_dirTSE = (D.f[dirTSE ])[ktse];
//       real f_dirTNW = (D.f[dirTNW ])[ktnw];
//       real f_dirBNE = (D.f[dirBNE ])[kbne];
//       real f_dirBSW = (D.f[dirBSW ])[kbsw];
//       real f_dirBSE = (D.f[dirBSE ])[kbse];
//       real f_dirBNW = (D.f[dirBNW ])[kbnw];


return {
        (D.f[dirE])[ke],
        (D.f[dirW])[kw],
        (D.f[dirN])[kn],
        (D.f[dirS])[ks],
        (D.f[dirT])[kt],
        (D.f[dirB])[kb],
        (D.f[dirNE])[kne],
        (D.f[dirSW])[ksw],
        (D.f[dirSE])[kse],
        (D.f[dirNW])[knw],
        (D.f[dirTE])[kte],
        (D.f[dirBW])[kbw],
        (D.f[dirBE])[kbe],
        (D.f[dirTW])[ktw],
        (D.f[dirTN])[ktn],
        (D.f[dirBS])[kbs],
        (D.f[dirBN])[kbn],
        (D.f[dirTS])[kts],
        (D.f[dirZERO])[kzero],
        (D.f[dirTNE ])[ktne],
        (D.f[dirTSW ])[ktsw],
        (D.f[dirTSE ])[ktse],
        (D.f[dirTNW ])[ktnw],
        (D.f[dirBNE ])[kbne],
        (D.f[dirBSW ])[kbsw],
        (D.f[dirBSE ])[kbse],
        (D.f[dirBNW ])[kbnw]
    };
}

////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMac27( real* vxD,
                                        real* vyD,
                                        real* vzD,
                                        real* rhoD,
                                        unsigned int* geoD,
                                        unsigned int* neighborX,
                                        unsigned int* neighborY,
                                        unsigned int* neighborZ,
                                        unsigned int size_Mat,
                                        real* DD,
                                        bool evenOrOdd)
{
   const unsigned int tx = threadIdx.x;    // Thread index = lokaler i index
   const unsigned int by = blockIdx.x;     // Block index x
   const unsigned int bz = blockIdx.y;     // Block index y
   const unsigned int x = tx + STARTOFFX;  // Globaler x-Index 
   const unsigned int y = by + STARTOFFY;  // Globaler y-Index 
   const unsigned int z = bz + STARTOFFZ;  // Globaler z-Index 

   const unsigned sizeX = blockDim.x;
   const unsigned sizeY = gridDim.x;
   const unsigned nx = sizeX + 2 * STARTOFFX;
   const unsigned ny = sizeY + 2 * STARTOFFY;

   const unsigned int k = nx*(ny*z + y) + x; // Zugriff auf arrays im device

   rhoD[k] = c0o1;
   vxD[k]  = c0o1;
   vyD[k]  = c0o1;
   vzD[k]  = c0o1;

   if(geoD[k] == GEO_FLUID)
   {
       const auto distribution = getDistribution(DD, size_Mat, evenOrOdd, k, neighborX, neighborY, neighborZ);

       rhoD[k] = LBM::getDensity(distribution.f);
       vxD[k] = LBM::getIncompVelocityX1(distribution.f);
       vyD[k] = LBM::getIncompVelocityX2(distribution.f);
       vzD[k] = LBM::getIncompVelocityX3(distribution.f);
   }
}





////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacSP27( real* vxD,
                                          real* vyD,
                                          real* vzD,
                                          real* rhoD,
                                          real* pressD,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD,
                                          bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
	  rhoD[k]   = c0o1;
	  vxD[k]    = c0o1;
	  vyD[k]    = c0o1;
	  vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vxD[k]     =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         vyD[k]     =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vzD[k]     =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         pressD[k]  =  ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        2.f*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        3.f*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1+c0o1*rhoD[k])) * c1o2+rhoD[k]; // times zero for incompressible case   
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5

      }
   }
}




























////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacCompSP27( real* vxD,
											  real* vyD,
											  real* vzD,
											  real* rhoD,
											  real* pressD,
											  unsigned int* geoD,
											  unsigned int* neighborX,
											  unsigned int* neighborY,
											  unsigned int* neighborZ,
											  unsigned int size_Mat,
											  real* DD,
											  bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
	  rhoD[k]   = c0o1;
	  vxD[k]    = c0o1;
	  vyD[k]    = c0o1;
	  vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID || geoD[k] == GEO_PM_0 || geoD[k] == GEO_PM_1 || geoD[k] == GEO_PM_2)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vxD[k]     =  ((D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
						(D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (c1o1 + rhoD[k]);

         vyD[k]     =  ((D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]) / (c1o1 + rhoD[k]);

         vzD[k]     =  ((D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (c1o1 + rhoD[k]);

         pressD[k]  =  ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        c2o1*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        c3o1*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1+rhoD[k])) * c1o2+rhoD[k]; // times zero for incompressible case   
         //achtung op hart gesetzt Annahme op = 1 ;                                                      ^^^^(1.0/op-0.5)=0.5

      }
   }
}



























////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacThS7( real* Conc,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD7,
                                          bool evenOrOdd)
{
   Distributions7 D7;
   if (evenOrOdd==true)
   {
      D7.f[0] = &DD7[0*size_Mat];
      D7.f[1] = &DD7[1*size_Mat];
      D7.f[2] = &DD7[2*size_Mat];
      D7.f[3] = &DD7[3*size_Mat];
      D7.f[4] = &DD7[4*size_Mat];
      D7.f[5] = &DD7[5*size_Mat];
      D7.f[6] = &DD7[6*size_Mat];
   } 
   else
   {
      D7.f[0] = &DD7[0*size_Mat];
      D7.f[2] = &DD7[1*size_Mat];
      D7.f[1] = &DD7[2*size_Mat];
      D7.f[4] = &DD7[3*size_Mat];
      D7.f[3] = &DD7[4*size_Mat];
      D7.f[6] = &DD7[5*size_Mat];
      D7.f[5] = &DD7[6*size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D7.f[1])[ke   ]+ (D7.f[2])[kw  ]+ 
                        (D7.f[3])[kn   ]+ (D7.f[4])[ks  ]+
                        (D7.f[5])[kt   ]+ (D7.f[6])[kb  ]+
                        (D7.f[0])[kzero];  
      }
   }
}





























////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void GetPlaneConcThS7(real* Conc,
								            int* kPC,
								            unsigned int numberOfPointskPC,
											unsigned int* geoD,
											unsigned int* neighborX,
											unsigned int* neighborY,
											unsigned int* neighborZ,
											unsigned int size_Mat,
											real* DD7,
											bool evenOrOdd)
{
   Distributions7 D7;
   if (evenOrOdd==true)
   {
      D7.f[0] = &DD7[0*size_Mat];
      D7.f[1] = &DD7[1*size_Mat];
      D7.f[2] = &DD7[2*size_Mat];
      D7.f[3] = &DD7[3*size_Mat];
      D7.f[4] = &DD7[4*size_Mat];
      D7.f[5] = &DD7[5*size_Mat];
      D7.f[6] = &DD7[6*size_Mat];
   } 
   else
   {
      D7.f[0] = &DD7[0*size_Mat];
      D7.f[2] = &DD7[1*size_Mat];
      D7.f[1] = &DD7[2*size_Mat];
      D7.f[4] = &DD7[3*size_Mat];
      D7.f[3] = &DD7[4*size_Mat];
      D7.f[6] = &DD7[5*size_Mat];
      D7.f[5] = &DD7[6*size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfPointskPC)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kPC[k];
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D7.f[1])[ke   ]+ (D7.f[2])[kw  ]+ 
                        (D7.f[3])[kn   ]+ (D7.f[4])[ks  ]+
                        (D7.f[5])[kt   ]+ (D7.f[6])[kb  ]+
                        (D7.f[0])[kzero];  
      }
   }
}




































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void GetPlaneConcThS27(real* Conc,
								             int* kPC,
								             unsigned int numberOfPointskPC,
											 unsigned int* geoD,
											 unsigned int* neighborX,
											 unsigned int* neighborY,
											 unsigned int* neighborZ,
											 unsigned int size_Mat,
											 real* DD27,
											 bool evenOrOdd)
{
   Distributions27 D27;
   if (evenOrOdd==true)
   {
      D27.f[dirE   ] = &DD27[dirE   *size_Mat];
      D27.f[dirW   ] = &DD27[dirW   *size_Mat];
      D27.f[dirN   ] = &DD27[dirN   *size_Mat];
      D27.f[dirS   ] = &DD27[dirS   *size_Mat];
      D27.f[dirT   ] = &DD27[dirT   *size_Mat];
      D27.f[dirB   ] = &DD27[dirB   *size_Mat];
      D27.f[dirNE  ] = &DD27[dirNE  *size_Mat];
      D27.f[dirSW  ] = &DD27[dirSW  *size_Mat];
      D27.f[dirSE  ] = &DD27[dirSE  *size_Mat];
      D27.f[dirNW  ] = &DD27[dirNW  *size_Mat];
      D27.f[dirTE  ] = &DD27[dirTE  *size_Mat];
      D27.f[dirBW  ] = &DD27[dirBW  *size_Mat];
      D27.f[dirBE  ] = &DD27[dirBE  *size_Mat];
      D27.f[dirTW  ] = &DD27[dirTW  *size_Mat];
      D27.f[dirTN  ] = &DD27[dirTN  *size_Mat];
      D27.f[dirBS  ] = &DD27[dirBS  *size_Mat];
      D27.f[dirBN  ] = &DD27[dirBN  *size_Mat];
      D27.f[dirTS  ] = &DD27[dirTS  *size_Mat];
      D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
      D27.f[dirTNE ] = &DD27[dirTNE *size_Mat];
      D27.f[dirTSW ] = &DD27[dirTSW *size_Mat];
      D27.f[dirTSE ] = &DD27[dirTSE *size_Mat];
      D27.f[dirTNW ] = &DD27[dirTNW *size_Mat];
      D27.f[dirBNE ] = &DD27[dirBNE *size_Mat];
      D27.f[dirBSW ] = &DD27[dirBSW *size_Mat];
      D27.f[dirBSE ] = &DD27[dirBSE *size_Mat];
      D27.f[dirBNW ] = &DD27[dirBNW *size_Mat];
   }
   else
   {
      D27.f[dirW   ] = &DD27[dirE   *size_Mat];
      D27.f[dirE   ] = &DD27[dirW   *size_Mat];
      D27.f[dirS   ] = &DD27[dirN   *size_Mat];
      D27.f[dirN   ] = &DD27[dirS   *size_Mat];
      D27.f[dirB   ] = &DD27[dirT   *size_Mat];
      D27.f[dirT   ] = &DD27[dirB   *size_Mat];
      D27.f[dirSW  ] = &DD27[dirNE  *size_Mat];
      D27.f[dirNE  ] = &DD27[dirSW  *size_Mat];
      D27.f[dirNW  ] = &DD27[dirSE  *size_Mat];
      D27.f[dirSE  ] = &DD27[dirNW  *size_Mat];
      D27.f[dirBW  ] = &DD27[dirTE  *size_Mat];
      D27.f[dirTE  ] = &DD27[dirBW  *size_Mat];
      D27.f[dirTW  ] = &DD27[dirBE  *size_Mat];
      D27.f[dirBE  ] = &DD27[dirTW  *size_Mat];
      D27.f[dirBS  ] = &DD27[dirTN  *size_Mat];
      D27.f[dirTN  ] = &DD27[dirBS  *size_Mat];
      D27.f[dirTS  ] = &DD27[dirBN  *size_Mat];
      D27.f[dirBN  ] = &DD27[dirTS  *size_Mat];
      D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
      D27.f[dirBSW ] = &DD27[dirTNE *size_Mat];
      D27.f[dirBNE ] = &DD27[dirTSW *size_Mat];
      D27.f[dirBNW ] = &DD27[dirTSE *size_Mat];
      D27.f[dirBSE ] = &DD27[dirTNW *size_Mat];
      D27.f[dirTSW ] = &DD27[dirBNE *size_Mat];
      D27.f[dirTNE ] = &DD27[dirBSW *size_Mat];
      D27.f[dirTNW ] = &DD27[dirBSE *size_Mat];
      D27.f[dirTSE ] = &DD27[dirBNW *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfPointskPC)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kPC[k];
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = kzero;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = kzero;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = kzero;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = kzero;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D27.f[dirE   ])[ke  ]+ (D27.f[dirW   ])[kw  ]+ 
                        (D27.f[dirN   ])[kn  ]+ (D27.f[dirS   ])[ks  ]+
                        (D27.f[dirT   ])[kt  ]+ (D27.f[dirB   ])[kb  ]+
                        (D27.f[dirNE  ])[kne ]+ (D27.f[dirSW  ])[ksw ]+
                        (D27.f[dirSE  ])[kse ]+ (D27.f[dirNW  ])[knw ]+
                        (D27.f[dirTE  ])[kte ]+ (D27.f[dirBW  ])[kbw ]+
                        (D27.f[dirBE  ])[kbe ]+ (D27.f[dirTW  ])[ktw ]+
                        (D27.f[dirTN  ])[ktn ]+ (D27.f[dirBS  ])[kbs ]+
                        (D27.f[dirBN  ])[kbn ]+ (D27.f[dirTS  ])[kts ]+
                        (D27.f[dirZERO])[kzero]+ 
                        (D27.f[dirTNE ])[ktne]+ (D27.f[dirTSW ])[ktsw]+
                        (D27.f[dirTSE ])[ktse]+ (D27.f[dirTNW ])[ktnw]+
                        (D27.f[dirBNE ])[kbne]+ (D27.f[dirBSW ])[kbsw]+
                        (D27.f[dirBSE ])[kbse]+ (D27.f[dirBNW ])[kbnw];
      }
   }   
}




































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacThS27(real* Conc,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD27,
                                          bool evenOrOdd)
{
   Distributions27 D27;
   if (evenOrOdd==true)
   {
      D27.f[dirE   ] = &DD27[dirE   *size_Mat];
      D27.f[dirW   ] = &DD27[dirW   *size_Mat];
      D27.f[dirN   ] = &DD27[dirN   *size_Mat];
      D27.f[dirS   ] = &DD27[dirS   *size_Mat];
      D27.f[dirT   ] = &DD27[dirT   *size_Mat];
      D27.f[dirB   ] = &DD27[dirB   *size_Mat];
      D27.f[dirNE  ] = &DD27[dirNE  *size_Mat];
      D27.f[dirSW  ] = &DD27[dirSW  *size_Mat];
      D27.f[dirSE  ] = &DD27[dirSE  *size_Mat];
      D27.f[dirNW  ] = &DD27[dirNW  *size_Mat];
      D27.f[dirTE  ] = &DD27[dirTE  *size_Mat];
      D27.f[dirBW  ] = &DD27[dirBW  *size_Mat];
      D27.f[dirBE  ] = &DD27[dirBE  *size_Mat];
      D27.f[dirTW  ] = &DD27[dirTW  *size_Mat];
      D27.f[dirTN  ] = &DD27[dirTN  *size_Mat];
      D27.f[dirBS  ] = &DD27[dirBS  *size_Mat];
      D27.f[dirBN  ] = &DD27[dirBN  *size_Mat];
      D27.f[dirTS  ] = &DD27[dirTS  *size_Mat];
      D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
      D27.f[dirTNE ] = &DD27[dirTNE *size_Mat];
      D27.f[dirTSW ] = &DD27[dirTSW *size_Mat];
      D27.f[dirTSE ] = &DD27[dirTSE *size_Mat];
      D27.f[dirTNW ] = &DD27[dirTNW *size_Mat];
      D27.f[dirBNE ] = &DD27[dirBNE *size_Mat];
      D27.f[dirBSW ] = &DD27[dirBSW *size_Mat];
      D27.f[dirBSE ] = &DD27[dirBSE *size_Mat];
      D27.f[dirBNW ] = &DD27[dirBNW *size_Mat];
   }
   else
   {
      D27.f[dirW   ] = &DD27[dirE   *size_Mat];
      D27.f[dirE   ] = &DD27[dirW   *size_Mat];
      D27.f[dirS   ] = &DD27[dirN   *size_Mat];
      D27.f[dirN   ] = &DD27[dirS   *size_Mat];
      D27.f[dirB   ] = &DD27[dirT   *size_Mat];
      D27.f[dirT   ] = &DD27[dirB   *size_Mat];
      D27.f[dirSW  ] = &DD27[dirNE  *size_Mat];
      D27.f[dirNE  ] = &DD27[dirSW  *size_Mat];
      D27.f[dirNW  ] = &DD27[dirSE  *size_Mat];
      D27.f[dirSE  ] = &DD27[dirNW  *size_Mat];
      D27.f[dirBW  ] = &DD27[dirTE  *size_Mat];
      D27.f[dirTE  ] = &DD27[dirBW  *size_Mat];
      D27.f[dirTW  ] = &DD27[dirBE  *size_Mat];
      D27.f[dirBE  ] = &DD27[dirTW  *size_Mat];
      D27.f[dirBS  ] = &DD27[dirTN  *size_Mat];
      D27.f[dirTN  ] = &DD27[dirBS  *size_Mat];
      D27.f[dirTS  ] = &DD27[dirBN  *size_Mat];
      D27.f[dirBN  ] = &DD27[dirTS  *size_Mat];
      D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
      D27.f[dirBSW ] = &DD27[dirTNE *size_Mat];
      D27.f[dirBNE ] = &DD27[dirTSW *size_Mat];
      D27.f[dirBNW ] = &DD27[dirTSE *size_Mat];
      D27.f[dirBSE ] = &DD27[dirTNW *size_Mat];
      D27.f[dirTSW ] = &DD27[dirBNE *size_Mat];
      D27.f[dirTNE ] = &DD27[dirBSW *size_Mat];
      D27.f[dirTNW ] = &DD27[dirBSE *size_Mat];
      D27.f[dirTSE ] = &DD27[dirBNW *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D27.f[dirE   ])[ke  ]+ (D27.f[dirW   ])[kw  ]+ 
                        (D27.f[dirN   ])[kn  ]+ (D27.f[dirS   ])[ks  ]+
                        (D27.f[dirT   ])[kt  ]+ (D27.f[dirB   ])[kb  ]+
                        (D27.f[dirNE  ])[kne ]+ (D27.f[dirSW  ])[ksw ]+
                        (D27.f[dirSE  ])[kse ]+ (D27.f[dirNW  ])[knw ]+
                        (D27.f[dirTE  ])[kte ]+ (D27.f[dirBW  ])[kbw ]+
                        (D27.f[dirBE  ])[kbe ]+ (D27.f[dirTW  ])[ktw ]+
                        (D27.f[dirTN  ])[ktn ]+ (D27.f[dirBS  ])[kbs ]+
                        (D27.f[dirBN  ])[kbn ]+ (D27.f[dirTS  ])[kts ]+
                        (D27.f[dirZERO])[kzero]+ 
                        (D27.f[dirTNE ])[ktne]+ (D27.f[dirTSW ])[ktsw]+
                        (D27.f[dirTSE ])[ktse]+ (D27.f[dirTNW ])[ktnw]+
                        (D27.f[dirBNE ])[kbne]+ (D27.f[dirBSW ])[kbsw]+
                        (D27.f[dirBSE ])[kbse]+ (D27.f[dirBNW ])[kbnw];
      }
   }   
}




















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMedSP27( real* vxD,
                                          real* vyD,
                                          real* vzD,
                                          real* rhoD,
                                          real* pressD,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD,
                                          bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
	  rhoD[k]   = c0o1;
	  vxD[k]    = c0o1;
	  vyD[k]    = c0o1;
	  vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]+
                        RHO;

         vxD[k]     =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]+
                        VX;

         vyD[k]     =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]+
                        VY;

         vzD[k]     =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]+
                        VZ;

         pressD[k]  =   ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        c2o1*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        c3o1*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1+rhoD[k])) * c1o2+rhoD[k]+
                        PRESS;    
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMedCompSP27( real* vxD,
											  real* vyD,
											  real* vzD,
											  real* rhoD,
											  real* pressD,
											  unsigned int* geoD,
											  unsigned int* neighborX,
											  unsigned int* neighborY,
											  unsigned int* neighborZ,
											  unsigned int size_Mat,
											  real* DD,
											  bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      //unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
	  rhoD[k]   = c0o1;
	  vxD[k]    = c0o1;
	  vyD[k]    = c0o1;
	  vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
		  real mfcbb = (D.f[dirE])[k];//[ke   ];
		  real mfabb = (D.f[dirW])[kw];//[kw   ];  
		  real mfbcb = (D.f[dirN])[k];//[kn   ];
		  real mfbab = (D.f[dirS])[ks];//[ks   ];  
		  real mfbbc = (D.f[dirT])[k];//[kt   ];
		  real mfbba = (D.f[dirB])[kb];//[kb   ];  
		  real mfccb = (D.f[dirNE])[k];//[kne  ];  
		  real mfaab = (D.f[dirSW])[ksw];//[ksw  ];
		  real mfcab = (D.f[dirSE])[ks];//[kse  ]; 
		  real mfacb = (D.f[dirNW])[kw];//[knw  ]; 
		  real mfcbc = (D.f[dirTE])[k];//[kte  ];  
		  real mfaba = (D.f[dirBW])[kbw];//[kbw  ];
		  real mfcba = (D.f[dirBE])[kb];//[kbe  ]; 
		  real mfabc = (D.f[dirTW])[kw];//[ktw  ]; 
		  real mfbcc = (D.f[dirTN])[k];//[ktn  ];  
		  real mfbaa = (D.f[dirBS])[kbs];//[kbs  ];
		  real mfbca = (D.f[dirBN])[kb];//[kbn  ]; 
		  real mfbac = (D.f[dirTS])[ks];//[kts  ]; 
		  real mfbbb = (D.f[dirZERO])[k];//[kzero];
		  real mfccc = (D.f[dirTNE])[k];//[ktne ]; 
		  real mfaac = (D.f[dirTSW])[ksw];//[ktsw ]; 
		  real mfcac = (D.f[dirTSE])[ks];//[ktse ];
		  real mfacc = (D.f[dirTNW])[kw];//[ktnw ];
		  real mfcca = (D.f[dirBNE])[kb];//[kbne ];
		  real mfaaa = (D.f[dirBSW])[kbsw];//[kbsw ];
		  real mfcaa = (D.f[dirBSE])[kbs];//[kbse ]; 
		  real mfaca = (D.f[dirBNW])[kbw];//[kbnw ]; 
		  ////////////////////////////////////////////////////////////////////////////////////
		  real drho = 
			  ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
			  (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
			  ((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;

		  real rho = c1o1 + drho;
		  
		  rhoD[k] = drho + RHO;

		  vxD[k] = 
			  (((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
			  (((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
			  (mfcbb - mfabb)) / rho) + VX;
		  vyD[k] = 
			  (((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
			  (((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
			  (mfbcb - mfbab)) / rho) + VY;
		  vzD[k] = 
			  (((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
			  (((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
			  (mfbbc - mfbba)) / rho) + VZ;

		  //rhoD[k] =
			 // (D.f[dirE])[ke] + (D.f[dirW])[kw] +
			 // (D.f[dirN])[kn] + (D.f[dirS])[ks] +
			 // (D.f[dirT])[kt] + (D.f[dirB])[kb] +
			 // (D.f[dirNE])[kne] + (D.f[dirSW])[ksw] +
			 // (D.f[dirSE])[kse] + (D.f[dirNW])[knw] +
			 // (D.f[dirTE])[kte] + (D.f[dirBW])[kbw] +
			 // (D.f[dirBE])[kbe] + (D.f[dirTW])[ktw] +
			 // (D.f[dirTN])[ktn] + (D.f[dirBS])[kbs] +
			 // (D.f[dirBN])[kbn] + (D.f[dirTS])[kts] +
			 // (D.f[dirZERO])[kzero] +
			 // (D.f[dirTNE])[ktne] + (D.f[dirTSW])[ktsw] +
			 // (D.f[dirTSE])[ktse] + (D.f[dirTNW])[ktnw] +
			 // (D.f[dirBNE])[kbne] + (D.f[dirBSW])[kbsw] +
			 // (D.f[dirBSE])[kbse] + (D.f[dirBNW])[kbnw];// +RHO;

    //     vxD[k] =  
			 //((D.f[dirE  ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
    //         (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
    //         (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
    //         (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
    //         (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
    //         (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
    //         (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
    //         (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
    //         (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
    //         VX;

    //     vyD[k] =  
			 //((D.f[dirN  ])[kn  ]- (D.f[dirS   ])[ks  ]+
    //         (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
    //         (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
    //         (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
    //         (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
    //         (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
    //         (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
    //         (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
    //         (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
    //         VY;

    //     vzD[k] =  
			 //((D.f[dirT  ])[kt  ]- (D.f[dirB   ])[kb  ]+
    //         (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
    //         (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
    //         (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
    //         (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
    //         (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
    //         (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
    //         (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
    //         (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
    //         VZ;

         pressD[k]  =  ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        c2o1*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        c3o1*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1+rhoD[k])) * c1o2+rhoD[k]+
                        PRESS;    
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMedCompAD27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	real* concD,
	unsigned int* geoD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	real* DD,
	real* DD_AD,
	bool evenOrOdd)
{
	Distributions27 D;
	if (evenOrOdd == true)
	{
		D.f[dirE] = &DD[dirE   *size_Mat];
		D.f[dirW] = &DD[dirW   *size_Mat];
		D.f[dirN] = &DD[dirN   *size_Mat];
		D.f[dirS] = &DD[dirS   *size_Mat];
		D.f[dirT] = &DD[dirT   *size_Mat];
		D.f[dirB] = &DD[dirB   *size_Mat];
		D.f[dirNE] = &DD[dirNE  *size_Mat];
		D.f[dirSW] = &DD[dirSW  *size_Mat];
		D.f[dirSE] = &DD[dirSE  *size_Mat];
		D.f[dirNW] = &DD[dirNW  *size_Mat];
		D.f[dirTE] = &DD[dirTE  *size_Mat];
		D.f[dirBW] = &DD[dirBW  *size_Mat];
		D.f[dirBE] = &DD[dirBE  *size_Mat];
		D.f[dirTW] = &DD[dirTW  *size_Mat];
		D.f[dirTN] = &DD[dirTN  *size_Mat];
		D.f[dirBS] = &DD[dirBS  *size_Mat];
		D.f[dirBN] = &DD[dirBN  *size_Mat];
		D.f[dirTS] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE] = &DD[dirTNE *size_Mat];
		D.f[dirTSW] = &DD[dirTSW *size_Mat];
		D.f[dirTSE] = &DD[dirTSE *size_Mat];
		D.f[dirTNW] = &DD[dirTNW *size_Mat];
		D.f[dirBNE] = &DD[dirBNE *size_Mat];
		D.f[dirBSW] = &DD[dirBSW *size_Mat];
		D.f[dirBSE] = &DD[dirBSE *size_Mat];
		D.f[dirBNW] = &DD[dirBNW *size_Mat];
	}
	else
	{
		D.f[dirW] = &DD[dirE   *size_Mat];
		D.f[dirE] = &DD[dirW   *size_Mat];
		D.f[dirS] = &DD[dirN   *size_Mat];
		D.f[dirN] = &DD[dirS   *size_Mat];
		D.f[dirB] = &DD[dirT   *size_Mat];
		D.f[dirT] = &DD[dirB   *size_Mat];
		D.f[dirSW] = &DD[dirNE  *size_Mat];
		D.f[dirNE] = &DD[dirSW  *size_Mat];
		D.f[dirNW] = &DD[dirSE  *size_Mat];
		D.f[dirSE] = &DD[dirNW  *size_Mat];
		D.f[dirBW] = &DD[dirTE  *size_Mat];
		D.f[dirTE] = &DD[dirBW  *size_Mat];
		D.f[dirTW] = &DD[dirBE  *size_Mat];
		D.f[dirBE] = &DD[dirTW  *size_Mat];
		D.f[dirBS] = &DD[dirTN  *size_Mat];
		D.f[dirTN] = &DD[dirBS  *size_Mat];
		D.f[dirTS] = &DD[dirBN  *size_Mat];
		D.f[dirBN] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE] = &DD[dirBSW *size_Mat];
		D.f[dirTSW] = &DD[dirBNE *size_Mat];
		D.f[dirTSE] = &DD[dirBNW *size_Mat];
		D.f[dirTNW] = &DD[dirBSE *size_Mat];
		D.f[dirBNE] = &DD[dirTSW *size_Mat];
		D.f[dirBSW] = &DD[dirTNE *size_Mat];
		D.f[dirBSE] = &DD[dirTNW *size_Mat];
		D.f[dirBNW] = &DD[dirTSE *size_Mat];
	}
	////////////////////////////////////////////////////////////////////////////////
	Distributions27 Dad;
	if (evenOrOdd == true)
	{
		Dad.f[dirE]    = &DD_AD[dirE   *size_Mat];
		Dad.f[dirW]    = &DD_AD[dirW   *size_Mat];
		Dad.f[dirN]    = &DD_AD[dirN   *size_Mat];
		Dad.f[dirS]    = &DD_AD[dirS   *size_Mat];
		Dad.f[dirT]    = &DD_AD[dirT   *size_Mat];
		Dad.f[dirB]    = &DD_AD[dirB   *size_Mat];
		Dad.f[dirNE]   = &DD_AD[dirNE  *size_Mat];
		Dad.f[dirSW]   = &DD_AD[dirSW  *size_Mat];
		Dad.f[dirSE]   = &DD_AD[dirSE  *size_Mat];
		Dad.f[dirNW]   = &DD_AD[dirNW  *size_Mat];
		Dad.f[dirTE]   = &DD_AD[dirTE  *size_Mat];
		Dad.f[dirBW]   = &DD_AD[dirBW  *size_Mat];
		Dad.f[dirBE]   = &DD_AD[dirBE  *size_Mat];
		Dad.f[dirTW]   = &DD_AD[dirTW  *size_Mat];
		Dad.f[dirTN]   = &DD_AD[dirTN  *size_Mat];
		Dad.f[dirBS]   = &DD_AD[dirBS  *size_Mat];
		Dad.f[dirBN]   = &DD_AD[dirBN  *size_Mat];
		Dad.f[dirTS]   = &DD_AD[dirTS  *size_Mat];
		Dad.f[dirZERO] = &DD_AD[dirZERO*size_Mat];
		Dad.f[dirTNE]  = &DD_AD[dirTNE *size_Mat];
		Dad.f[dirTSW]  = &DD_AD[dirTSW *size_Mat];
		Dad.f[dirTSE]  = &DD_AD[dirTSE *size_Mat];
		Dad.f[dirTNW]  = &DD_AD[dirTNW *size_Mat];
		Dad.f[dirBNE]  = &DD_AD[dirBNE *size_Mat];
		Dad.f[dirBSW]  = &DD_AD[dirBSW *size_Mat];
		Dad.f[dirBSE]  = &DD_AD[dirBSE *size_Mat];
		Dad.f[dirBNW]  = &DD_AD[dirBNW *size_Mat];
	}						
	else					
	{						
		Dad.f[dirW]    = &DD_AD[dirE   *size_Mat];
		Dad.f[dirE]    = &DD_AD[dirW   *size_Mat];
		Dad.f[dirS]    = &DD_AD[dirN   *size_Mat];
		Dad.f[dirN]    = &DD_AD[dirS   *size_Mat];
		Dad.f[dirB]    = &DD_AD[dirT   *size_Mat];
		Dad.f[dirT]    = &DD_AD[dirB   *size_Mat];
		Dad.f[dirSW]   = &DD_AD[dirNE  *size_Mat];
		Dad.f[dirNE]   = &DD_AD[dirSW  *size_Mat];
		Dad.f[dirNW]   = &DD_AD[dirSE  *size_Mat];
		Dad.f[dirSE]   = &DD_AD[dirNW  *size_Mat];
		Dad.f[dirBW]   = &DD_AD[dirTE  *size_Mat];
		Dad.f[dirTE]   = &DD_AD[dirBW  *size_Mat];
		Dad.f[dirTW]   = &DD_AD[dirBE  *size_Mat];
		Dad.f[dirBE]   = &DD_AD[dirTW  *size_Mat];
		Dad.f[dirBS]   = &DD_AD[dirTN  *size_Mat];
		Dad.f[dirTN]   = &DD_AD[dirBS  *size_Mat];
		Dad.f[dirTS]   = &DD_AD[dirBN  *size_Mat];
		Dad.f[dirBN]   = &DD_AD[dirTS  *size_Mat];
		Dad.f[dirZERO] = &DD_AD[dirZERO*size_Mat];
		Dad.f[dirTNE]  = &DD_AD[dirBSW *size_Mat];
		Dad.f[dirTSW]  = &DD_AD[dirBNE *size_Mat];
		Dad.f[dirTSE]  = &DD_AD[dirBNW *size_Mat];
		Dad.f[dirTNW]  = &DD_AD[dirBSE *size_Mat];
		Dad.f[dirBNE]  = &DD_AD[dirTSW *size_Mat];
		Dad.f[dirBSW]  = &DD_AD[dirTNE *size_Mat];
		Dad.f[dirBSE]  = &DD_AD[dirTNW *size_Mat];
		Dad.f[dirBNW]  = &DD_AD[dirTSE *size_Mat];
	}
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k < size_Mat)
	{
		//////////////////////////////////////////////////////////////////////////
		//index
		//unsigned int kzero = k;
		unsigned int ke = k;
		unsigned int kw = neighborX[k];
		unsigned int kn = k;
		unsigned int ks = neighborY[k];
		unsigned int kt = k;
		unsigned int kb = neighborZ[k];
		unsigned int ksw = neighborY[kw];
		unsigned int kne = k;
		unsigned int kse = ks;
		unsigned int knw = kw;
		unsigned int kbw = neighborZ[kw];
		unsigned int kte = k;
		unsigned int kbe = kb;
		unsigned int ktw = kw;
		unsigned int kbs = neighborZ[ks];
		unsigned int ktn = k;
		unsigned int kbn = kb;
		unsigned int kts = ks;
		unsigned int ktse = ks;
		unsigned int kbnw = kbw;
		unsigned int ktnw = kw;
		unsigned int kbse = kbs;
		unsigned int ktsw = ksw;
		unsigned int kbne = kb;
		unsigned int ktne = k;
		unsigned int kbsw = neighborZ[ksw];
		//////////////////////////////////////////////////////////////////////////
		real CONC  = concD[k];
		real PRESS = pressD[k];
		real RHO   = rhoD[k];
		real VX    = vxD[k];
		real VY    = vyD[k];
		real VZ    = vzD[k];
		//////////////////////////////////////////////////////////////////////////
		concD[k] = c0o1;
		pressD[k] = c0o1;
		rhoD[k] = c0o1;
		vxD[k] = c0o1;
		vyD[k] = c0o1;
		vzD[k] = c0o1;

		if (geoD[k] == GEO_FLUID)
		{
			real mfcbb = (D.f[dirE])[k];//[ke   ];
			real mfabb = (D.f[dirW])[kw];//[kw   ];  
			real mfbcb = (D.f[dirN])[k];//[kn   ];
			real mfbab = (D.f[dirS])[ks];//[ks   ];  
			real mfbbc = (D.f[dirT])[k];//[kt   ];
			real mfbba = (D.f[dirB])[kb];//[kb   ];  
			real mfccb = (D.f[dirNE])[k];//[kne  ];  
			real mfaab = (D.f[dirSW])[ksw];//[ksw  ];
			real mfcab = (D.f[dirSE])[ks];//[kse  ]; 
			real mfacb = (D.f[dirNW])[kw];//[knw  ]; 
			real mfcbc = (D.f[dirTE])[k];//[kte  ];  
			real mfaba = (D.f[dirBW])[kbw];//[kbw  ];
			real mfcba = (D.f[dirBE])[kb];//[kbe  ]; 
			real mfabc = (D.f[dirTW])[kw];//[ktw  ]; 
			real mfbcc = (D.f[dirTN])[k];//[ktn  ];  
			real mfbaa = (D.f[dirBS])[kbs];//[kbs  ];
			real mfbca = (D.f[dirBN])[kb];//[kbn  ]; 
			real mfbac = (D.f[dirTS])[ks];//[kts  ]; 
			real mfbbb = (D.f[dirZERO])[k];//[kzero];
			real mfccc = (D.f[dirTNE])[k];//[ktne ]; 
			real mfaac = (D.f[dirTSW])[ksw];//[ktsw ]; 
			real mfcac = (D.f[dirTSE])[ks];//[ktse ];
			real mfacc = (D.f[dirTNW])[kw];//[ktnw ];
			real mfcca = (D.f[dirBNE])[kb];//[kbne ];
			real mfaaa = (D.f[dirBSW])[kbsw];//[kbsw ];
			real mfcaa = (D.f[dirBSE])[kbs];//[kbse ]; 
			real mfaca = (D.f[dirBNW])[kbw];//[kbnw ]; 
			////////////////////////////////////////////////////////////////////////////////////
			real drho =
				((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				 (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				  ((mfabb + mfcbb) + (mfbab + mfbcb)  +  (mfbba + mfbbc))) + mfbbb;
			real rho = c1o1 + drho;
			////////////////////////////////////////////////////////////////////////////////////

			rhoD[k] = drho + RHO;

			vxD[k] =
				(((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
				(((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
					(mfcbb - mfabb)) / rho) + VX;
			
			vyD[k] =
				(((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
				(((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
					(mfbcb - mfbab)) / rho) + VY;
			
			vzD[k] =
				(((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
				(((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
					(mfbbc - mfbba)) / rho) + VZ;

			pressD[k] = 
				((D.f[dirE])[ke] + (D.f[dirW])[kw] +
				 (D.f[dirN])[kn] + (D.f[dirS])[ks] +
				 (D.f[dirT])[kt] + (D.f[dirB])[kb] +
				 c2o1*(
				 (D.f[dirNE])[kne] + (D.f[dirSW])[ksw] +
				 (D.f[dirSE])[kse] + (D.f[dirNW])[knw] +
				 (D.f[dirTE])[kte] + (D.f[dirBW])[kbw] +
				 (D.f[dirBE])[kbe] + (D.f[dirTW])[ktw] +
				 (D.f[dirTN])[ktn] + (D.f[dirBS])[kbs] +
				 (D.f[dirBN])[kbn] + (D.f[dirTS])[kts]) +
				 c3o1*(
				 (D.f[dirTNE])[ktne] + (D.f[dirTSW])[ktsw] +
				 (D.f[dirTSE])[ktse] + (D.f[dirTNW])[ktnw] +
				 (D.f[dirBNE])[kbne] + (D.f[dirBSW])[kbsw] +
				 (D.f[dirBSE])[kbse] + (D.f[dirBNW])[kbnw]) -
				 rhoD[k] - (vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (c1o1 + rhoD[k])) * c1o2 + rhoD[k] +
				 PRESS;
				 //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5
			//////////////////////////////////////////////////////////////////////////
			mfcbb = (Dad.f[dirE   ])[k   ];
			mfabb = (Dad.f[dirW   ])[kw  ];
			mfbcb = (Dad.f[dirN   ])[k   ];
			mfbab = (Dad.f[dirS   ])[ks  ];
			mfbbc = (Dad.f[dirT   ])[k   ];
			mfbba = (Dad.f[dirB   ])[kb  ];
			mfccb = (Dad.f[dirNE  ])[k   ];
			mfaab = (Dad.f[dirSW  ])[ksw ];
			mfcab = (Dad.f[dirSE  ])[ks  ];
			mfacb = (Dad.f[dirNW  ])[kw  ];
			mfcbc = (Dad.f[dirTE  ])[k   ];
			mfaba = (Dad.f[dirBW  ])[kbw ];
			mfcba = (Dad.f[dirBE  ])[kb  ];
			mfabc = (Dad.f[dirTW  ])[kw  ];
			mfbcc = (Dad.f[dirTN  ])[k   ];
			mfbaa = (Dad.f[dirBS  ])[kbs ];
			mfbca = (Dad.f[dirBN  ])[kb  ];
			mfbac = (Dad.f[dirTS  ])[ks  ];
			mfbbb = (Dad.f[dirZERO])[k   ];
			mfccc = (Dad.f[dirTNE ])[k   ];
			mfaac = (Dad.f[dirTSW ])[ksw ];
			mfcac = (Dad.f[dirTSE ])[ks  ];
			mfacc = (Dad.f[dirTNW ])[kw  ];
			mfcca = (Dad.f[dirBNE ])[kb  ];
			mfaaa = (Dad.f[dirBSW ])[kbsw];
			mfcaa = (Dad.f[dirBSE ])[kbs ];
			mfaca = (Dad.f[dirBNW ])[kbw ];
			//////////////////////////////////////////////////////////////////////////
			concD[k] = 
				((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa)   + (mfaac + mfcca))) +
				 (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba)   + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				  ((mfabb + mfcbb) + (mfbab + mfbcb)  +  (mfbba + mfbbc))) +  mfbbb + CONC;
		}
	}
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacMedSP27( real* vxD,
                                             real* vyD,
                                             real* vzD,
                                             real* rhoD,
                                             real* pressD,
                                             unsigned int* geoD,
                                             unsigned int* neighborX,
                                             unsigned int* neighborY,
                                             unsigned int* neighborZ,
                                             unsigned int tdiff,
                                             unsigned int size_Mat,
                                             bool evenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = c0o1;
      rhoD[k]   = c0o1;
      vxD[k]    = c0o1;
      vyD[k]    = c0o1;
      vzD[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   RHO   / tdiff;
         vxD[k]     =   VX    / tdiff;
         vyD[k]     =   VY    / tdiff;
         vzD[k]     =   VZ    / tdiff;
         pressD[k]  =   PRESS / tdiff;    
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBResetMedianValuesSP27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	unsigned int size_Mat,
	bool evenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k<size_Mat)
	{
		//////////////////////////////////////////////////////////////////////////
		pressD[k] = c0o1;
		rhoD[k] = c0o1;
		vxD[k] = c0o1;
		vyD[k] = c0o1;
		vzD[k] = c0o1;
	}
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBResetMedianValuesAD27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	real* concD,
	unsigned int size_Mat,
	bool evenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k < size_Mat)
	{
		//////////////////////////////////////////////////////////////////////////
		concD[k]  = c0o1;
		pressD[k] = c0o1;
		rhoD[k]   = c0o1;
		vxD[k]    = c0o1;
		vyD[k]    = c0o1;
		vzD[k]    = c0o1;
	}
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMeasurePoints( real* vxMP,
												real* vyMP,
												real* vzMP,
												real* rhoMP,
												unsigned int* kMP,
												unsigned int numberOfPointskMP,
												unsigned int MPClockCycle,
												unsigned int t,
												unsigned int* geoD,
												unsigned int* neighborX,
												unsigned int* neighborY,
												unsigned int* neighborZ,
												unsigned int size_Mat,
												real* DD,
												bool evenOrOdd)
{
	Distributions27 D;
	if (evenOrOdd==true)
	{
		D.f[dirE   ] = &DD[dirE   *size_Mat];
		D.f[dirW   ] = &DD[dirW   *size_Mat];
		D.f[dirN   ] = &DD[dirN   *size_Mat];
		D.f[dirS   ] = &DD[dirS   *size_Mat];
		D.f[dirT   ] = &DD[dirT   *size_Mat];
		D.f[dirB   ] = &DD[dirB   *size_Mat];
		D.f[dirNE  ] = &DD[dirNE  *size_Mat];
		D.f[dirSW  ] = &DD[dirSW  *size_Mat];
		D.f[dirSE  ] = &DD[dirSE  *size_Mat];
		D.f[dirNW  ] = &DD[dirNW  *size_Mat];
		D.f[dirTE  ] = &DD[dirTE  *size_Mat];
		D.f[dirBW  ] = &DD[dirBW  *size_Mat];
		D.f[dirBE  ] = &DD[dirBE  *size_Mat];
		D.f[dirTW  ] = &DD[dirTW  *size_Mat];
		D.f[dirTN  ] = &DD[dirTN  *size_Mat];
		D.f[dirBS  ] = &DD[dirBS  *size_Mat];
		D.f[dirBN  ] = &DD[dirBN  *size_Mat];
		D.f[dirTS  ] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE ] = &DD[dirTNE *size_Mat];
		D.f[dirTSW ] = &DD[dirTSW *size_Mat];
		D.f[dirTSE ] = &DD[dirTSE *size_Mat];
		D.f[dirTNW ] = &DD[dirTNW *size_Mat];
		D.f[dirBNE ] = &DD[dirBNE *size_Mat];
		D.f[dirBSW ] = &DD[dirBSW *size_Mat];
		D.f[dirBSE ] = &DD[dirBSE *size_Mat];
		D.f[dirBNW ] = &DD[dirBNW *size_Mat];
	} 
	else
	{
		D.f[dirW   ] = &DD[dirE   *size_Mat];
		D.f[dirE   ] = &DD[dirW   *size_Mat];
		D.f[dirS   ] = &DD[dirN   *size_Mat];
		D.f[dirN   ] = &DD[dirS   *size_Mat];
		D.f[dirB   ] = &DD[dirT   *size_Mat];
		D.f[dirT   ] = &DD[dirB   *size_Mat];
		D.f[dirSW  ] = &DD[dirNE  *size_Mat];
		D.f[dirNE  ] = &DD[dirSW  *size_Mat];
		D.f[dirNW  ] = &DD[dirSE  *size_Mat];
		D.f[dirSE  ] = &DD[dirNW  *size_Mat];
		D.f[dirBW  ] = &DD[dirTE  *size_Mat];
		D.f[dirTE  ] = &DD[dirBW  *size_Mat];
		D.f[dirTW  ] = &DD[dirBE  *size_Mat];
		D.f[dirBE  ] = &DD[dirTW  *size_Mat];
		D.f[dirBS  ] = &DD[dirTN  *size_Mat];
		D.f[dirTN  ] = &DD[dirBS  *size_Mat];
		D.f[dirTS  ] = &DD[dirBN  *size_Mat];
		D.f[dirBN  ] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE ] = &DD[dirBSW *size_Mat];
		D.f[dirTSW ] = &DD[dirBNE *size_Mat];
		D.f[dirTSE ] = &DD[dirBNW *size_Mat];
		D.f[dirTNW ] = &DD[dirBSE *size_Mat];
		D.f[dirBNE ] = &DD[dirTSW *size_Mat];
		D.f[dirBSW ] = &DD[dirTNE *size_Mat];
		D.f[dirBSE ] = &DD[dirTNW *size_Mat];
		D.f[dirBNW ] = &DD[dirTSE *size_Mat];
	}
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if(k<numberOfPointskMP)
	{
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kMP[k];//k;
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = kzero;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = kzero;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = kzero;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = kzero;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
	  unsigned int kMac = k*MPClockCycle + t;
	  //////////////////////////////////////////////////////////////////////////

      if(geoD[kzero] == GEO_FLUID)
      {
         rhoMP[kMac]=   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vxMP[kMac] =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         vyMP[kMac] =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vzMP[kMac] =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBSetOutputWallVelocitySP27( real* vxD,
														real* vyD,
														real* vzD,
														real* vxWall,
														real* vyWall,
														real* vzWall,
														int numberOfWallNodes, 
														int* kWallNodes, 
														real* rhoD,
														real* pressD,
														unsigned int* geoD,
														unsigned int* neighborX,
														unsigned int* neighborY,
														unsigned int* neighborZ,
														unsigned int size_Mat,
														real* DD,
														bool evenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfWallNodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KWN  = kWallNodes[k];
      //////////////////////////////////////////////////////////////////////////
      vxD[KWN] = 0.0;//vxWall[k];
      vyD[KWN] = 0.0;//vyWall[k];
      vzD[KWN] = 0.0;//vzWall[k];
   }
}





























