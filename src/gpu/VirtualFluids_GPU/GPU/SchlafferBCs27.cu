#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h"
#include "lbm/constants/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;

// TODO: https://git.rz.tu-bs.de/irmb/VirtualFluids_dev/-/issues/29
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void PressSchlaff27(real* rhoBC,
                                          real* DD,
                                          real* vx0,
                                          real* vy0,
                                          real* vz0,
                                          real* deltaVz0,
                                          int* k_Q,
                                          int* k_N,
                                          int numberOfBCnodes,
                                          real om1,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned long long numberOfLBnodes,
                                          bool isEvenTimestep)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index
   const unsigned  y = blockIdx.x;   // Globaler y-Index
   const unsigned  z = blockIdx.y;   // Globaler z-Index

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      Distributions27 D;
      if (isEvenTimestep==true)
      {
         D.f[DIR_P00   ] = &DD[DIR_P00   *numberOfLBnodes];
         D.f[DIR_M00   ] = &DD[DIR_M00   *numberOfLBnodes];
         D.f[DIR_0P0   ] = &DD[DIR_0P0   *numberOfLBnodes];
         D.f[DIR_0M0   ] = &DD[DIR_0M0   *numberOfLBnodes];
         D.f[DIR_00P   ] = &DD[DIR_00P   *numberOfLBnodes];
         D.f[DIR_00M   ] = &DD[DIR_00M   *numberOfLBnodes];
         D.f[DIR_PP0  ] = &DD[DIR_PP0  *numberOfLBnodes];
         D.f[DIR_MM0  ] = &DD[DIR_MM0  *numberOfLBnodes];
         D.f[DIR_PM0  ] = &DD[DIR_PM0  *numberOfLBnodes];
         D.f[DIR_MP0  ] = &DD[DIR_MP0  *numberOfLBnodes];
         D.f[DIR_P0P  ] = &DD[DIR_P0P  *numberOfLBnodes];
         D.f[DIR_M0M  ] = &DD[DIR_M0M  *numberOfLBnodes];
         D.f[DIR_P0M  ] = &DD[DIR_P0M  *numberOfLBnodes];
         D.f[DIR_M0P  ] = &DD[DIR_M0P  *numberOfLBnodes];
         D.f[DIR_0PP  ] = &DD[DIR_0PP  *numberOfLBnodes];
         D.f[DIR_0MM  ] = &DD[DIR_0MM  *numberOfLBnodes];
         D.f[DIR_0PM  ] = &DD[DIR_0PM  *numberOfLBnodes];
         D.f[DIR_0MP  ] = &DD[DIR_0MP  *numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000*numberOfLBnodes];
         D.f[DIR_PPP ] = &DD[DIR_PPP *numberOfLBnodes];
         D.f[DIR_MMP ] = &DD[DIR_MMP *numberOfLBnodes];
         D.f[DIR_PMP ] = &DD[DIR_PMP *numberOfLBnodes];
         D.f[DIR_MPP ] = &DD[DIR_MPP *numberOfLBnodes];
         D.f[DIR_PPM ] = &DD[DIR_PPM *numberOfLBnodes];
         D.f[DIR_MMM ] = &DD[DIR_MMM *numberOfLBnodes];
         D.f[DIR_PMM ] = &DD[DIR_PMM *numberOfLBnodes];
         D.f[DIR_MPM ] = &DD[DIR_MPM *numberOfLBnodes];
      }
      else
      {
         D.f[DIR_M00   ] = &DD[DIR_P00   *numberOfLBnodes];
         D.f[DIR_P00   ] = &DD[DIR_M00   *numberOfLBnodes];
         D.f[DIR_0M0   ] = &DD[DIR_0P0   *numberOfLBnodes];
         D.f[DIR_0P0   ] = &DD[DIR_0M0   *numberOfLBnodes];
         D.f[DIR_00M   ] = &DD[DIR_00P   *numberOfLBnodes];
         D.f[DIR_00P   ] = &DD[DIR_00M   *numberOfLBnodes];
         D.f[DIR_MM0  ] = &DD[DIR_PP0  *numberOfLBnodes];
         D.f[DIR_PP0  ] = &DD[DIR_MM0  *numberOfLBnodes];
         D.f[DIR_MP0  ] = &DD[DIR_PM0  *numberOfLBnodes];
         D.f[DIR_PM0  ] = &DD[DIR_MP0  *numberOfLBnodes];
         D.f[DIR_M0M  ] = &DD[DIR_P0P  *numberOfLBnodes];
         D.f[DIR_P0P  ] = &DD[DIR_M0M  *numberOfLBnodes];
         D.f[DIR_M0P  ] = &DD[DIR_P0M  *numberOfLBnodes];
         D.f[DIR_P0M  ] = &DD[DIR_M0P  *numberOfLBnodes];
         D.f[DIR_0MM  ] = &DD[DIR_0PP  *numberOfLBnodes];
         D.f[DIR_0PP  ] = &DD[DIR_0MM  *numberOfLBnodes];
         D.f[DIR_0MP  ] = &DD[DIR_0PM  *numberOfLBnodes];
         D.f[DIR_0PM  ] = &DD[DIR_0MP  *numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000*numberOfLBnodes];
         D.f[DIR_PPP ] = &DD[DIR_MMM *numberOfLBnodes];
         D.f[DIR_MMP ] = &DD[DIR_PPM *numberOfLBnodes];
         D.f[DIR_PMP ] = &DD[DIR_MPM *numberOfLBnodes];
         D.f[DIR_MPP ] = &DD[DIR_PMM *numberOfLBnodes];
         D.f[DIR_PPM ] = &DD[DIR_MMP *numberOfLBnodes];
         D.f[DIR_MMM ] = &DD[DIR_PPP *numberOfLBnodes];
         D.f[DIR_PMM ] = &DD[DIR_MPP *numberOfLBnodes];
         D.f[DIR_MPM ] = &DD[DIR_PMP *numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real        f1_E,f1_W,f1_N,f1_S,f1_T,f1_B,f1_NE,f1_SW,f1_SE,f1_NW,f1_TE,f1_BW,f1_BE,f1_TW,f1_TN,f1_BS,f1_BN,f1_TS,f1_ZERO,f1_TNE,f1_TSW,f1_TSE,f1_TNW,f1_BNE,f1_BSW,f1_BSE,f1_BNW;

      f1_E    = (D.f[DIR_P00   ])[ke   ];
      f1_W    = (D.f[DIR_M00   ])[kw   ];
      f1_N    = (D.f[DIR_0P0   ])[kn   ];
      f1_S    = (D.f[DIR_0M0   ])[ks   ];
      f1_T    = (D.f[DIR_00P   ])[kt   ];
      f1_B    = (D.f[DIR_00M   ])[kb   ];
      f1_NE   = (D.f[DIR_PP0  ])[kne  ];
      f1_SW   = (D.f[DIR_MM0  ])[ksw  ];
      f1_SE   = (D.f[DIR_PM0  ])[kse  ];
      f1_NW   = (D.f[DIR_MP0  ])[knw  ];
      f1_TE   = (D.f[DIR_P0P  ])[kte  ];
      f1_BW   = (D.f[DIR_M0M  ])[kbw  ];
      f1_BE   = (D.f[DIR_P0M  ])[kbe  ];
      f1_TW   = (D.f[DIR_M0P  ])[ktw  ];
      f1_TN   = (D.f[DIR_0PP  ])[ktn  ];
      f1_BS   = (D.f[DIR_0MM  ])[kbs  ];
      f1_BN   = (D.f[DIR_0PM  ])[kbn  ];
      f1_TS   = (D.f[DIR_0MP  ])[kts  ];
      f1_ZERO = (D.f[DIR_000])[kzero];
      f1_TNE  = (D.f[DIR_PPP ])[ktne ];
      f1_TSW  = (D.f[DIR_MMP ])[ktsw ];
      f1_TSE  = (D.f[DIR_PMP ])[ktse ];
      f1_TNW  = (D.f[DIR_MPP ])[ktnw ];
      f1_BNE  = (D.f[DIR_PPM ])[kbne ];
      f1_BSW  = (D.f[DIR_MMM ])[kbsw ];
      f1_BSE  = (D.f[DIR_PMM ])[kbse ];
      f1_BNW  = (D.f[DIR_MPM ])[kbnw ];
      //////////////////////////////////////////////////////////////////////////
      real cs       = c1o1/sqrt(c3o1);
      real csp1     = cs + c1o1;
      real csp1Sq  = (c1o1 + cs)*(c1o1 + cs);
      real relFac   = c21o20; // 0.9...1.0
      //////////////////////////////////////////////////////////////////////////
      // For adaption:
      //     Pressure limits with rho0 = 1:
      //      2.2e-10 ~  0.94 dB
      //      6.2e-10 ~   9.9 dB
      //      6.2e-9  ~  29.9 dB
      //      2.0e-7  ~  60.1 dB   /Vel
      //      2.0e-5  ~ 100.1 dB   /press
      const double dPlimit  = Op0000002;
      const double dRlimit  = dPlimit * c3o1;// three = c1oCs2;
      const double uSlimit  = dRlimit * c1o1;// one = c1oRho0;
      //////////////////////////////////////////////////////////////////////////
      real VX = vx0[k];
      real VY = vy0[k];
      real VZ = vz0[k];
      //////////////////////////////////////////////////////////////////////////

      real temp = c2o1*(f1_TNE + f1_TSE + f1_TSW + f1_TNW) + c2o1*(f1_TE + f1_TW + f1_TN + f1_TS) + f1_NE + f1_SW + f1_SE + f1_NW + c2o1*f1_T + f1_E + f1_W + f1_N + f1_S + f1_ZERO;

      real vs_z = relFac * (VZ+cs) * ( csp1 - sqrt(csp1Sq + c2o1*VZ - c2o1*temp) );    //old =  relFac * cs * ( csp1 - sqrt(csp1Sq + two*VZ - two*temp) );

      // 3. Compute density of compensated velocity:
      real tempDeltaV = deltaVz0[k];
      real rholoc = temp - c1o1 * (VZ + tempDeltaV + vs_z);

      // 4. Compute density deviation:
      real drho = rholoc - rhoBC[k];

      // 5. Adapt Speed:
      real dv = tempDeltaV + vs_z;

      if( drho > dRlimit) {
         VZ += dv + uSlimit;
         tempDeltaV += uSlimit;
      }
      else if( drho < -dRlimit) {
         VZ += dv - uSlimit;
         tempDeltaV -= uSlimit;
      }
      else {
         VZ += dv + drho;
         tempDeltaV += drho;
      }

      //VZ = vz0[k] + vs_z;
      // 6. Set unknown distributions:
      f1_B   = f1_T   - c4o9  * VZ;
      f1_BW  = f1_TE  - c1o9  * (VX + VZ);
      f1_BE  = f1_TW  + c1o9  * (VX - VZ);
      f1_BS  = f1_TN  - c1o9  * (VY + VZ);
      f1_BN  = f1_TS  + c1o9  * (VY - VZ);
      f1_BSW = f1_TNE - c1o36 * (VX + VY + VZ);
      f1_BNW = f1_TSE - c1o36 * (VX - VY + VZ);
      f1_BNE = f1_TSW + c1o36 * (VX + VY - VZ);
      f1_BSE = f1_TNW + c1o36 * (VX - VY - VZ);

      deltaVz0[k] = tempDeltaV;

      (D.f[DIR_00M   ])[kb   ] = f1_B   ;
      (D.f[DIR_M0M  ])[kbw  ] = f1_BW  ;
      (D.f[DIR_P0M  ])[kbe  ] = f1_BE  ;
      (D.f[DIR_0MM  ])[kbs  ] = f1_BS  ;
      (D.f[DIR_0PM  ])[kbn  ] = f1_BN  ;
      (D.f[DIR_PPM ])[kbne ] = f1_BNE ;
      (D.f[DIR_MMM ])[kbsw ] = f1_BSW ;
      (D.f[DIR_PMM ])[kbse ] = f1_BSE ;
      (D.f[DIR_MPM ])[kbnw ] = f1_BNW ;
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////






































// TODO: https://git.rz.tu-bs.de/irmb/VirtualFluids_dev/-/issues/29
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void VelSchlaff27(  int t,
                                          real* DD,
                                          real* vz0,
                                          real* deltaVz0,
                                          int* k_Q,
                                          int* k_N,
                                          int numberOfBCnodes,
                                          real om1,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned long long numberOfLBnodes,
                                          bool isEvenTimestep)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index
   const unsigned  y = blockIdx.x;   // Globaler y-Index
   const unsigned  z = blockIdx.y;   // Globaler z-Index

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      Distributions27 D;
      if (isEvenTimestep==true)
      {
         D.f[DIR_P00   ] = &DD[DIR_P00   *numberOfLBnodes];
         D.f[DIR_M00   ] = &DD[DIR_M00   *numberOfLBnodes];
         D.f[DIR_0P0   ] = &DD[DIR_0P0   *numberOfLBnodes];
         D.f[DIR_0M0   ] = &DD[DIR_0M0   *numberOfLBnodes];
         D.f[DIR_00P   ] = &DD[DIR_00P   *numberOfLBnodes];
         D.f[DIR_00M   ] = &DD[DIR_00M   *numberOfLBnodes];
         D.f[DIR_PP0  ] = &DD[DIR_PP0  *numberOfLBnodes];
         D.f[DIR_MM0  ] = &DD[DIR_MM0  *numberOfLBnodes];
         D.f[DIR_PM0  ] = &DD[DIR_PM0  *numberOfLBnodes];
         D.f[DIR_MP0  ] = &DD[DIR_MP0  *numberOfLBnodes];
         D.f[DIR_P0P  ] = &DD[DIR_P0P  *numberOfLBnodes];
         D.f[DIR_M0M  ] = &DD[DIR_M0M  *numberOfLBnodes];
         D.f[DIR_P0M  ] = &DD[DIR_P0M  *numberOfLBnodes];
         D.f[DIR_M0P  ] = &DD[DIR_M0P  *numberOfLBnodes];
         D.f[DIR_0PP  ] = &DD[DIR_0PP  *numberOfLBnodes];
         D.f[DIR_0MM  ] = &DD[DIR_0MM  *numberOfLBnodes];
         D.f[DIR_0PM  ] = &DD[DIR_0PM  *numberOfLBnodes];
         D.f[DIR_0MP  ] = &DD[DIR_0MP  *numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000*numberOfLBnodes];
         D.f[DIR_PPP ] = &DD[DIR_PPP *numberOfLBnodes];
         D.f[DIR_MMP ] = &DD[DIR_MMP *numberOfLBnodes];
         D.f[DIR_PMP ] = &DD[DIR_PMP *numberOfLBnodes];
         D.f[DIR_MPP ] = &DD[DIR_MPP *numberOfLBnodes];
         D.f[DIR_PPM ] = &DD[DIR_PPM *numberOfLBnodes];
         D.f[DIR_MMM ] = &DD[DIR_MMM *numberOfLBnodes];
         D.f[DIR_PMM ] = &DD[DIR_PMM *numberOfLBnodes];
         D.f[DIR_MPM ] = &DD[DIR_MPM *numberOfLBnodes];
      }
      else
      {
         D.f[DIR_M00   ] = &DD[DIR_P00   *numberOfLBnodes];
         D.f[DIR_P00   ] = &DD[DIR_M00   *numberOfLBnodes];
         D.f[DIR_0M0   ] = &DD[DIR_0P0   *numberOfLBnodes];
         D.f[DIR_0P0   ] = &DD[DIR_0M0   *numberOfLBnodes];
         D.f[DIR_00M   ] = &DD[DIR_00P   *numberOfLBnodes];
         D.f[DIR_00P   ] = &DD[DIR_00M   *numberOfLBnodes];
         D.f[DIR_MM0  ] = &DD[DIR_PP0  *numberOfLBnodes];
         D.f[DIR_PP0  ] = &DD[DIR_MM0  *numberOfLBnodes];
         D.f[DIR_MP0  ] = &DD[DIR_PM0  *numberOfLBnodes];
         D.f[DIR_PM0  ] = &DD[DIR_MP0  *numberOfLBnodes];
         D.f[DIR_M0M  ] = &DD[DIR_P0P  *numberOfLBnodes];
         D.f[DIR_P0P  ] = &DD[DIR_M0M  *numberOfLBnodes];
         D.f[DIR_M0P  ] = &DD[DIR_P0M  *numberOfLBnodes];
         D.f[DIR_P0M  ] = &DD[DIR_M0P  *numberOfLBnodes];
         D.f[DIR_0MM  ] = &DD[DIR_0PP  *numberOfLBnodes];
         D.f[DIR_0PP  ] = &DD[DIR_0MM  *numberOfLBnodes];
         D.f[DIR_0MP  ] = &DD[DIR_0PM  *numberOfLBnodes];
         D.f[DIR_0PM  ] = &DD[DIR_0MP  *numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000*numberOfLBnodes];
         D.f[DIR_PPP ] = &DD[DIR_MMM *numberOfLBnodes];
         D.f[DIR_MMP ] = &DD[DIR_PPM *numberOfLBnodes];
         D.f[DIR_PMP ] = &DD[DIR_MPM *numberOfLBnodes];
         D.f[DIR_MPP ] = &DD[DIR_PMM *numberOfLBnodes];
         D.f[DIR_PPM ] = &DD[DIR_MMP *numberOfLBnodes];
         D.f[DIR_MMM ] = &DD[DIR_PPP *numberOfLBnodes];
         D.f[DIR_PMM ] = &DD[DIR_MPP *numberOfLBnodes];
         D.f[DIR_MPM ] = &DD[DIR_PMP *numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real        f1_E,f1_W,f1_N,f1_S,f1_T,f1_B,f1_NE,f1_SW,f1_SE,f1_NW,f1_TE,f1_BW,f1_BE,f1_TW,f1_TN,f1_BS,f1_BN,f1_TS,f1_ZERO,
                     f1_TNE,f1_TSW,f1_TSE,f1_TNW,f1_BNE,f1_BSW,f1_BSE,f1_BNW;

      f1_E    = (D.f[DIR_P00   ])[ke   ];
      f1_W    = (D.f[DIR_M00   ])[kw   ];
      f1_N    = (D.f[DIR_0P0   ])[kn   ];
      f1_S    = (D.f[DIR_0M0   ])[ks   ];
      f1_T    = (D.f[DIR_00P   ])[kt   ];
      f1_B    = (D.f[DIR_00M   ])[kb   ];
      f1_NE   = (D.f[DIR_PP0  ])[kne  ];
      f1_SW   = (D.f[DIR_MM0  ])[ksw  ];
      f1_SE   = (D.f[DIR_PM0  ])[kse  ];
      f1_NW   = (D.f[DIR_MP0  ])[knw  ];
      f1_TE   = (D.f[DIR_P0P  ])[kte  ];
      f1_BW   = (D.f[DIR_M0M  ])[kbw  ];
      f1_BE   = (D.f[DIR_P0M  ])[kbe  ];
      f1_TW   = (D.f[DIR_M0P  ])[ktw  ];
      f1_TN   = (D.f[DIR_0PP  ])[ktn  ];
      f1_BS   = (D.f[DIR_0MM  ])[kbs  ];
      f1_BN   = (D.f[DIR_0PM  ])[kbn  ];
      f1_TS   = (D.f[DIR_0MP  ])[kts  ];
      f1_ZERO = (D.f[DIR_000])[kzero];
      f1_TNE  = (D.f[DIR_PPP ])[ktne ];
      f1_TSW  = (D.f[DIR_MMP ])[ktsw ];
      f1_TSE  = (D.f[DIR_PMP ])[ktse ];
      f1_TNW  = (D.f[DIR_MPP ])[ktnw ];
      f1_BNE  = (D.f[DIR_PPM ])[kbne ];
      f1_BSW  = (D.f[DIR_MMM ])[kbsw ];
      f1_BSE  = (D.f[DIR_PMM ])[kbse ];
      f1_BNW  = (D.f[DIR_MPM ])[kbnw ];
      //f1_W    = (D.f[DIR_P00   ])[ke   ];
      //f1_E    = (D.f[DIR_M00   ])[kw   ];
      //f1_S    = (D.f[DIR_0P0   ])[kn   ];
      //f1_N    = (D.f[DIR_0M0   ])[ks   ];
      //f1_B    = (D.f[DIR_00P   ])[kt   ];
      //f1_T    = (D.f[DIR_00M   ])[kb   ];
      //f1_SW   = (D.f[DIR_PP0  ])[kne  ];
      //f1_NE   = (D.f[DIR_MM0  ])[ksw  ];
      //f1_NW   = (D.f[DIR_PM0  ])[kse  ];
      //f1_SE   = (D.f[DIR_MP0  ])[knw  ];
      //f1_BW   = (D.f[DIR_P0P  ])[kte  ];
      //f1_TE   = (D.f[DIR_M0M  ])[kbw  ];
      //f1_TW   = (D.f[DIR_P0M  ])[kbe  ];
      //f1_BE   = (D.f[DIR_M0P  ])[ktw  ];
      //f1_BS   = (D.f[DIR_0PP  ])[ktn  ];
      //f1_TN   = (D.f[DIR_0MM  ])[kbs  ];
      //f1_TS   = (D.f[DIR_0PM  ])[kbn  ];
      //f1_BN   = (D.f[DIR_0MP  ])[kts  ];
      //f1_ZERO = (D.f[DIR_000])[kzero];
      //f1_BSW  = (D.f[DIR_PPP ])[ktne ];
      //f1_BNE  = (D.f[DIR_MMP ])[ktsw ];
      //f1_BNW  = (D.f[DIR_PMP ])[ktse ];
      //f1_BSE  = (D.f[DIR_MPP ])[ktnw ];
      //f1_TSW  = (D.f[DIR_PPM ])[kbne ];
      //f1_TNE  = (D.f[DIR_MMM ])[kbsw ];
      //f1_TNW  = (D.f[DIR_PMM ])[kbse ];
      //f1_TSE  = (D.f[DIR_MPM ])[kbnw ];
      //////////////////////////////////////////////////////////////////////////
      real cs       = c1o1/sqrt(c3o1);
      real csp1     = cs + c1o1;
      real csp1Sq  = (c1o1 + cs)*(c1o1 + cs);
      real relFac   = c19o20; // 0.9...1.0
      //////////////////////////////////////////////////////////////////////////
      // For adaption:
      //     Pressure limits with rho0 = 1:
      //      2.2e-10 ~  0.94 dB
      //      6.2e-10 ~   9.9 dB
      //      6.2e-9  ~  29.9 dB
      //      2.0e-7  ~  60.1 dB   /Vel
      //      2.0e-5  ~ 100.1 dB   /press
      real uSlimit  = Op0000002;
      //////////////////////////////////////////////////////////////////////////
      real VX = c0o1;
      real VY = c0o1;
      real VZ = vz0[k];
      //////////////////////////////////////////////////////////////////////////
      real temp = f1_ZERO + f1_E + f1_W + f1_N + f1_S + f1_NE + f1_SW + f1_SE + f1_NW + c2o1*(f1_B + f1_BE + f1_BW + f1_BN + f1_BS + f1_BNE + f1_BSE + f1_BSW + f1_BNW);
      //real temp = f1_ZERO + f1_E + f1_W + f1_N + f1_S + f1_NE + f1_SW + f1_SE + f1_NW + two*(f1_T + f1_TE + f1_TW + f1_TN + f1_TS + f1_TNE + f1_TSE + f1_TSW + f1_TNW);
      ////real temp2= c1mcsSq + two*VZ - two*temp;
      real vs_z;
      //if (t < 5)
      //{
      //   vs_z = zero;
      //}
      //else
      //{
         vs_z = relFac * (cs-VZ) * ( sqrt(csp1Sq - c2o1*VZ - c2o1*temp) - csp1 );         //old = relFac * cs * ( sqrt(csp1Sq - two*VZ - two*temp) - csp1 );
      //}

      // 3. Adapt Speed:
      real tempDeltaV = deltaVz0[k];
      real dv = tempDeltaV + vs_z;

      if( dv > uSlimit) {
         VZ  += dv - uSlimit;
         tempDeltaV -= uSlimit;
      }
      else if( dv < -uSlimit) {
         VZ  += dv + uSlimit;
         tempDeltaV += uSlimit;
      }
      else {
         tempDeltaV = -vs_z;
      }

      //VZ = vz0[k]+vs_z;
      // 4. Set unknown distributions:
      //f1_B   = f1_T   - c4o9  * VZ;
      //f1_BW  = f1_TE  - c1o9  * (VX + VZ);
      //f1_BE  = f1_TW  + c1o9  * (VX - VZ);
      //f1_BS  = f1_TN  - c1o9  * (VY + VZ);
      //f1_BN  = f1_TS  + c1o9  * (VY - VZ);
      //f1_BSW = f1_TNE - c1o36 * (VX + VY + VZ);
      //f1_BNW = f1_TSE - c1o36 * (VX - VY + VZ);
      //f1_BNE = f1_TSW + c1o36 * (VX + VY - VZ);
      //f1_BSE = f1_TNW + c1o36 * (VX - VY - VZ);

      f1_T   = f1_B   + c4o9  * VZ;
      f1_TE  = f1_BW  + c1o9  * (VX + VZ);
      f1_TW  = f1_BE  - c1o9  * (VX - VZ);
      f1_TN  = f1_BS  + c1o9  * (VY + VZ);
      f1_TS  = f1_BN  - c1o9  * (VY - VZ);
      f1_TNE = f1_BSW + c1o36 * (VX + VY + VZ);
      f1_TSE = f1_BNW + c1o36 * (VX - VY + VZ);
      f1_TSW = f1_BNE - c1o36 * (VX + VY - VZ);
      f1_TNW = f1_BSE - c1o36 * (VX - VY - VZ);

      deltaVz0[k] = tempDeltaV;
      (D.f[DIR_00P   ])[kt   ] = f1_T  ;
      (D.f[DIR_P0P  ])[kte  ] = f1_TE ;
      (D.f[DIR_M0P  ])[ktw  ] = f1_TW ;
      (D.f[DIR_0PP  ])[ktn  ] = f1_TN ;
      (D.f[DIR_0MP  ])[kts  ] = f1_TS ;
      (D.f[DIR_PPP ])[ktne ] = f1_TNE;
      (D.f[DIR_MMP ])[ktsw ] = f1_TSW;
      (D.f[DIR_PMP ])[ktse ] = f1_TSE;
      (D.f[DIR_MPP ])[ktnw ] = f1_TNW;

      //(D.f[DIR_00M   ])[kb   ] = f1_B   ;
      //(D.f[DIR_M0M  ])[kbw  ] = f1_BW  ;
      //(D.f[DIR_P0M  ])[kbe  ] = f1_BE  ;
      //(D.f[DIR_0MM  ])[kbs  ] = f1_BS  ;
      //(D.f[DIR_0PM  ])[kbn  ] = f1_BN  ;
      //(D.f[DIR_PPM ])[kbne ] = f1_BNE ;
      //(D.f[DIR_MMM ])[kbsw ] = f1_BSW ;
      //(D.f[DIR_PMM ])[kbse ] = f1_BSE ;
      //(D.f[DIR_MPM ])[kbnw ] = f1_BNW ;


      //(D.f[DIR_00P   ])[kt   ] = f1_B  ;
      //(D.f[DIR_P0P  ])[kte  ] = f1_BW ;
      //(D.f[DIR_M0P  ])[ktw  ] = f1_BE ;
      //(D.f[DIR_0PP  ])[ktn  ] = f1_BS ;
      //(D.f[DIR_0MP  ])[kts  ] = f1_BN ;
      //(D.f[DIR_PPP ])[ktne ] = f1_BSW;
      //(D.f[DIR_MMP ])[ktsw ] = f1_BNE;
      //(D.f[DIR_PMP ])[ktse ] = f1_BNW;
      //(D.f[DIR_MPP ])[ktnw ] = f1_BSE;

      //(D.f[DIR_P00   ])[ke   ] = f1_W   -c2over27*drho1;
      //(D.f[DIR_M00   ])[kw   ] = f1_E   -c2over27*drho1;
      //(D.f[DIR_0P0   ])[kn   ] = f1_S   -c2over27*drho1;
      //(D.f[DIR_0M0   ])[ks   ] = f1_N   -c2over27*drho1;
      //(D.f[DIR_00P   ])[kt   ] = f1_B   -c2over27*drho1;
      //(D.f[DIR_00M   ])[kb   ] = f1_T   -c2over27*drho1;
      //(D.f[DIR_PP0  ])[kne  ] = f1_SW  -c1over54*drho1;
      //(D.f[DIR_MM0  ])[ksw  ] = f1_NE  -c1over54*drho1;
      //(D.f[DIR_PM0  ])[kse  ] = f1_NW  -c1over54*drho1;
      //(D.f[DIR_MP0  ])[knw  ] = f1_SE  -c1over54*drho1;
      //(D.f[DIR_P0P  ])[kte  ] = f1_BW  -c1over54*drho1;
      //(D.f[DIR_M0M  ])[kbw  ] = f1_TE  -c1over54*drho1;
      //(D.f[DIR_P0M  ])[kbe  ] = f1_TW  -c1over54*drho1;
      //(D.f[DIR_M0P  ])[ktw  ] = f1_BE  -c1over54*drho1;
      //(D.f[DIR_0PP  ])[ktn  ] = f1_BS  -c1over54*drho1;
      //(D.f[DIR_0MM  ])[kbs  ] = f1_TN  -c1over54*drho1;
      //(D.f[DIR_0PM  ])[kbn  ] = f1_TS  -c1over54*drho1;
      //(D.f[DIR_0MP  ])[kts  ] = f1_BN  -c1over54*drho1;
      //(D.f[DIR_000])[kzero] = f1_ZERO-c8over27*drho1;
      //(D.f[DIR_PPP ])[ktne ] = f1_BSW -c1over216*drho1;
      //(D.f[DIR_MMP ])[ktsw ] = f1_BNE -c1over216*drho1;
      //(D.f[DIR_PMP ])[ktse ] = f1_BNW -c1over216*drho1;
      //(D.f[DIR_MPP ])[ktnw ] = f1_BSE -c1over216*drho1;
      //(D.f[DIR_PPM ])[kbne ] = f1_TSW -c1over216*drho1;
      //(D.f[DIR_MMM ])[kbsw ] = f1_TNE -c1over216*drho1;
      //(D.f[DIR_PMM ])[kbse ] = f1_TNW -c1over216*drho1;
      //(D.f[DIR_MPM ])[kbnw ] = f1_TSE -c1over216*drho1;
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////





