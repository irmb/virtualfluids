#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;

////////////////////////////////////////////////////////////////////////////////
__global__ void LBCalc2ndMomentsIncompSP27(  real* kxyFromfcNEQ,
														real* kyzFromfcNEQ,
														real* kxzFromfcNEQ,
														real* kxxMyyFromfcNEQ,
														real* kxxMzzFromfcNEQ,
														unsigned int* geoD,
														unsigned int* neighborX,
														unsigned int* neighborY,
														unsigned int* neighborZ,
														unsigned int size_Mat,
														real* DD,
														bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[E   ] = &DD[E   *size_Mat];
      D.f[W   ] = &DD[W   *size_Mat];
      D.f[N   ] = &DD[N   *size_Mat];
      D.f[S   ] = &DD[S   *size_Mat];
      D.f[T   ] = &DD[T   *size_Mat];
      D.f[B   ] = &DD[B   *size_Mat];
      D.f[NE  ] = &DD[NE  *size_Mat];
      D.f[SW  ] = &DD[SW  *size_Mat];
      D.f[SE  ] = &DD[SE  *size_Mat];
      D.f[NW  ] = &DD[NW  *size_Mat];
      D.f[TE  ] = &DD[TE  *size_Mat];
      D.f[BW  ] = &DD[BW  *size_Mat];
      D.f[BE  ] = &DD[BE  *size_Mat];
      D.f[TW  ] = &DD[TW  *size_Mat];
      D.f[TN  ] = &DD[TN  *size_Mat];
      D.f[BS  ] = &DD[BS  *size_Mat];
      D.f[BN  ] = &DD[BN  *size_Mat];
      D.f[TS  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[TNE *size_Mat];
      D.f[TSW ] = &DD[TSW *size_Mat];
      D.f[TSE ] = &DD[TSE *size_Mat];
      D.f[TNW ] = &DD[TNW *size_Mat];
      D.f[BNE ] = &DD[BNE *size_Mat];
      D.f[BSW ] = &DD[BSW *size_Mat];
      D.f[BSE ] = &DD[BSE *size_Mat];
      D.f[BNW ] = &DD[BNW *size_Mat];
   } 
   else
   {
      D.f[W   ] = &DD[E   *size_Mat];
      D.f[E   ] = &DD[W   *size_Mat];
      D.f[S   ] = &DD[N   *size_Mat];
      D.f[N   ] = &DD[S   *size_Mat];
      D.f[B   ] = &DD[T   *size_Mat];
      D.f[T   ] = &DD[B   *size_Mat];
      D.f[SW  ] = &DD[NE  *size_Mat];
      D.f[NE  ] = &DD[SW  *size_Mat];
      D.f[NW  ] = &DD[SE  *size_Mat];
      D.f[SE  ] = &DD[NW  *size_Mat];
      D.f[BW  ] = &DD[TE  *size_Mat];
      D.f[TE  ] = &DD[BW  *size_Mat];
      D.f[TW  ] = &DD[BE  *size_Mat];
      D.f[BE  ] = &DD[TW  *size_Mat];
      D.f[BS  ] = &DD[TN  *size_Mat];
      D.f[TN  ] = &DD[BS  *size_Mat];
      D.f[TS  ] = &DD[BN  *size_Mat];
      D.f[BN  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[BSW *size_Mat];
      D.f[TSW ] = &DD[BNE *size_Mat];
      D.f[TSE ] = &DD[BNW *size_Mat];
      D.f[TNW ] = &DD[BSE *size_Mat];
      D.f[BNE ] = &DD[TSW *size_Mat];
      D.f[BSW ] = &DD[TNE *size_Mat];
      D.f[BSE ] = &DD[TNW *size_Mat];
      D.f[BNW ] = &DD[TSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k < size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      //unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      real        f_E,f_W,f_N,f_S,f_T,f_B,f_NE,f_SW,f_SE,f_NW,f_TE,f_BW,f_BE,f_TW,f_TN,f_BS,f_BN,f_TS,/*f_ZERO,*/f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;
	  f_E    = (D.f[E   ])[ke   ];
	  f_W    = (D.f[W   ])[kw   ];
	  f_N    = (D.f[N   ])[kn   ];
	  f_S    = (D.f[S   ])[ks   ];
	  f_T    = (D.f[T   ])[kt   ];
	  f_B    = (D.f[B   ])[kb   ];
	  f_NE   = (D.f[NE  ])[kne  ];
	  f_SW   = (D.f[SW  ])[ksw  ];
	  f_SE   = (D.f[SE  ])[kse  ];
	  f_NW   = (D.f[NW  ])[knw  ];
	  f_TE   = (D.f[TE  ])[kte  ];
	  f_BW   = (D.f[BW  ])[kbw  ];
	  f_BE   = (D.f[BE  ])[kbe  ];
	  f_TW   = (D.f[TW  ])[ktw  ];
	  f_TN   = (D.f[TN  ])[ktn  ];
	  f_BS   = (D.f[BS  ])[kbs  ];
	  f_BN   = (D.f[BN  ])[kbn  ];
	  f_TS   = (D.f[TS  ])[kts  ];
	  //f_ZERO = (D.f[REST])[kzero];
	  f_TNE  = (D.f[TNE ])[ktne ];
	  f_TSW  = (D.f[TSW ])[ktsw ];
	  f_TSE  = (D.f[TSE ])[ktse ];
	  f_TNW  = (D.f[TNW ])[ktnw ];
	  f_BNE  = (D.f[BNE ])[kbne ];
	  f_BSW  = (D.f[BSW ])[kbsw ];
	  f_BSE  = (D.f[BSE ])[kbse ];
	  f_BNW  = (D.f[BNW ])[kbnw ];
      //////////////////////////////////////////////////////////////////////////
	  real vx1, vx2, vx3;
      kxyFromfcNEQ[k]       = c0o1;
	  kyzFromfcNEQ[k]       = c0o1;
	  kxzFromfcNEQ[k]       = c0o1;
	  kxxMyyFromfcNEQ[k]    = c0o1;
	  kxxMzzFromfcNEQ[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
		  vx1                = ((f_TNE-f_BSW)+(f_BSE-f_TNW)+(f_BNE-f_TSW)+(f_TSE-f_BNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W);
		  vx2                = ((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S);
		  vx3                = ((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSW-f_BNE)+(f_TSE-f_BNW)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B);
		  kxyFromfcNEQ[k]    = -c3o1 *(f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE-(vx1*vx2));
		  kyzFromfcNEQ[k]    = -c3o1 *(f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW-(vx2*vx3));
		  kxzFromfcNEQ[k]    = -c3o1 *(f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE-(vx1*vx3));
		  kxxMyyFromfcNEQ[k] = -c3o2 * (f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE-(vx1*vx1-vx2*vx2));		//all E+W minus all N+S (no combinations of xy left)
		  kxxMzzFromfcNEQ[k] = -c3o2 * (f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE-(vx1*vx1-vx3*vx3));		//all E+W minus all T+B (no combinations of xz left)
      }
   }
}
////////////////////////////////////////////////////////////////////////////////































////////////////////////////////////////////////////////////////////////////////
__global__ void LBCalc2ndMomentsCompSP27(real* kxyFromfcNEQ,
													real* kyzFromfcNEQ,
													real* kxzFromfcNEQ,
													real* kxxMyyFromfcNEQ,
													real* kxxMzzFromfcNEQ,
													unsigned int* geoD,
													unsigned int* neighborX,
													unsigned int* neighborY,
													unsigned int* neighborZ,
													unsigned int size_Mat,
													real* DD,
													bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[E   ] = &DD[E   *size_Mat];
      D.f[W   ] = &DD[W   *size_Mat];
      D.f[N   ] = &DD[N   *size_Mat];
      D.f[S   ] = &DD[S   *size_Mat];
      D.f[T   ] = &DD[T   *size_Mat];
      D.f[B   ] = &DD[B   *size_Mat];
      D.f[NE  ] = &DD[NE  *size_Mat];
      D.f[SW  ] = &DD[SW  *size_Mat];
      D.f[SE  ] = &DD[SE  *size_Mat];
      D.f[NW  ] = &DD[NW  *size_Mat];
      D.f[TE  ] = &DD[TE  *size_Mat];
      D.f[BW  ] = &DD[BW  *size_Mat];
      D.f[BE  ] = &DD[BE  *size_Mat];
      D.f[TW  ] = &DD[TW  *size_Mat];
      D.f[TN  ] = &DD[TN  *size_Mat];
      D.f[BS  ] = &DD[BS  *size_Mat];
      D.f[BN  ] = &DD[BN  *size_Mat];
      D.f[TS  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[TNE *size_Mat];
      D.f[TSW ] = &DD[TSW *size_Mat];
      D.f[TSE ] = &DD[TSE *size_Mat];
      D.f[TNW ] = &DD[TNW *size_Mat];
      D.f[BNE ] = &DD[BNE *size_Mat];
      D.f[BSW ] = &DD[BSW *size_Mat];
      D.f[BSE ] = &DD[BSE *size_Mat];
      D.f[BNW ] = &DD[BNW *size_Mat];
   } 
   else
   {
      D.f[W   ] = &DD[E   *size_Mat];
      D.f[E   ] = &DD[W   *size_Mat];
      D.f[S   ] = &DD[N   *size_Mat];
      D.f[N   ] = &DD[S   *size_Mat];
      D.f[B   ] = &DD[T   *size_Mat];
      D.f[T   ] = &DD[B   *size_Mat];
      D.f[SW  ] = &DD[NE  *size_Mat];
      D.f[NE  ] = &DD[SW  *size_Mat];
      D.f[NW  ] = &DD[SE  *size_Mat];
      D.f[SE  ] = &DD[NW  *size_Mat];
      D.f[BW  ] = &DD[TE  *size_Mat];
      D.f[TE  ] = &DD[BW  *size_Mat];
      D.f[TW  ] = &DD[BE  *size_Mat];
      D.f[BE  ] = &DD[TW  *size_Mat];
      D.f[BS  ] = &DD[TN  *size_Mat];
      D.f[TN  ] = &DD[BS  *size_Mat];
      D.f[TS  ] = &DD[BN  *size_Mat];
      D.f[BN  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[BSW *size_Mat];
      D.f[TSW ] = &DD[BNE *size_Mat];
      D.f[TSE ] = &DD[BNW *size_Mat];
      D.f[TNW ] = &DD[BSE *size_Mat];
      D.f[BNE ] = &DD[TSW *size_Mat];
      D.f[BSW ] = &DD[TNE *size_Mat];
      D.f[BSE ] = &DD[TNW *size_Mat];
      D.f[BNW ] = &DD[TSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k < size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      real f_ZERO;
      real        f_E,f_W,f_N,f_S,f_T,f_B,f_NE,f_SW,f_SE,f_NW,f_TE,f_BW,f_BE,f_TW,f_TN,f_BS,f_BN,f_TS,f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;
	  f_E    = (D.f[E   ])[ke   ];
	  f_W    = (D.f[W   ])[kw   ];
	  f_N    = (D.f[N   ])[kn   ];
	  f_S    = (D.f[S   ])[ks   ];
	  f_T    = (D.f[T   ])[kt   ];
	  f_B    = (D.f[B   ])[kb   ];
	  f_NE   = (D.f[NE  ])[kne  ];
	  f_SW   = (D.f[SW  ])[ksw  ];
	  f_SE   = (D.f[SE  ])[kse  ];
	  f_NW   = (D.f[NW  ])[knw  ];
	  f_TE   = (D.f[TE  ])[kte  ];
	  f_BW   = (D.f[BW  ])[kbw  ];
	  f_BE   = (D.f[BE  ])[kbe  ];
	  f_TW   = (D.f[TW  ])[ktw  ];
	  f_TN   = (D.f[TN  ])[ktn  ];
	  f_BS   = (D.f[BS  ])[kbs  ];
	  f_BN   = (D.f[BN  ])[kbn  ];
	  f_TS   = (D.f[TS  ])[kts  ];
	  f_ZERO = (D.f[REST])[kzero];
	  f_TNE  = (D.f[TNE ])[ktne ];
	  f_TSW  = (D.f[TSW ])[ktsw ];
	  f_TSE  = (D.f[TSE ])[ktse ];
	  f_TNW  = (D.f[TNW ])[ktnw ];
	  f_BNE  = (D.f[BNE ])[kbne ];
	  f_BSW  = (D.f[BSW ])[kbsw ];
	  f_BSE  = (D.f[BSE ])[kbse ];
	  f_BNW  = (D.f[BNW ])[kbnw ];
      //////////////////////////////////////////////////////////////////////////
	  real drho;
	  real vx1, vx2, vx3, rho;
      kxyFromfcNEQ[k]       = c0o1;
	  kyzFromfcNEQ[k]       = c0o1;
	  kxzFromfcNEQ[k]       = c0o1;
	  kxxMyyFromfcNEQ[k]    = c0o1;
	  kxxMzzFromfcNEQ[k]    = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
          drho               = ((f_TNE+f_BSW)+(f_BSE+f_TNW)+(f_BNE+f_TSW)+(f_TSE+f_BNW)) +
		 					   ((f_NE+f_SW)+(f_TE+f_BW)+(f_SE+f_NW)+(f_BE+f_TW)+(f_BN+f_TS)+(f_TN+f_BS)) +
		 					   ((f_E-f_W) + (f_N-f_S) + (f_T-f_B)) + f_ZERO;
		  rho                = drho + c1o1;
		  vx1                = ((f_TNE-f_BSW)+(f_BSE-f_TNW)+(f_BNE-f_TSW)+(f_TSE-f_BNW)) + (((f_NE-f_SW)+(f_TE-f_BW))+((f_SE-f_NW)+(f_BE-f_TW))) + (f_E-f_W) / rho;
		  vx2                = ((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_BNE-f_TSW)+(f_BNW-f_TSE)) + (((f_NE-f_SW)+(f_TN-f_BS))+((f_BN-f_TS)+(f_NW-f_SE))) + (f_N-f_S) / rho;
		  vx3                = ((f_TNE-f_BSW)+(f_TNW-f_BSE)+(f_TSW-f_BNE)+(f_TSE-f_BNW)) + (((f_TE-f_BW)+(f_TN-f_BS))+((f_TW-f_BE)+(f_TS-f_BN))) + (f_T-f_B) / rho;
		  kxyFromfcNEQ[k]    = -c3o1 *(f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE-(vx1*vx2));
		  kyzFromfcNEQ[k]    = -c3o1 *(f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW-(vx2*vx3));
		  kxzFromfcNEQ[k]    = -c3o1 *(f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE-(vx1*vx3));
		  kxxMyyFromfcNEQ[k] = -c3o2 * (f_BW+f_W+f_TW-f_BS-f_S-f_TS-f_BN-f_N-f_TN+f_BE+f_E+f_TE-(vx1*vx1-vx2*vx2));		//all E+W minus all N+S (no combinations of xy left)
		  kxxMzzFromfcNEQ[k] = -c3o2 * (f_SW+f_W+f_NW-f_BS-f_TS-f_B-f_T-f_BN-f_TN+f_SE+f_E+f_NE-(vx1*vx1-vx3*vx3));		//all E+W minus all T+B (no combinations of xz left)
      }
   }
}
////////////////////////////////////////////////////////////////////////////////































////////////////////////////////////////////////////////////////////////////////
__global__ void LBCalc3rdMomentsIncompSP27(  real* CUMbbb,
														real* CUMabc,
														real* CUMbac,
														real* CUMbca,
														real* CUMcba,
														real* CUMacb,
														real* CUMcab,
														unsigned int* bcMatD,
														unsigned int* neighborX,
														unsigned int* neighborY,
														unsigned int* neighborZ,
														real* DDStart,
														int size_Mat,
														bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if(k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if( (BC != GEO_SOLID) && (BC != GEO_VOID) )
		{
			Distributions27 D;
			if (EvenOrOdd==true)
			{
				D.f[E   ] = &DDStart[E   *size_Mat];
				D.f[W   ] = &DDStart[W   *size_Mat];
				D.f[N   ] = &DDStart[N   *size_Mat];
				D.f[S   ] = &DDStart[S   *size_Mat];
				D.f[T   ] = &DDStart[T   *size_Mat];
				D.f[B   ] = &DDStart[B   *size_Mat];
				D.f[NE  ] = &DDStart[NE  *size_Mat];
				D.f[SW  ] = &DDStart[SW  *size_Mat];
				D.f[SE  ] = &DDStart[SE  *size_Mat];
				D.f[NW  ] = &DDStart[NW  *size_Mat];
				D.f[TE  ] = &DDStart[TE  *size_Mat];
				D.f[BW  ] = &DDStart[BW  *size_Mat];
				D.f[BE  ] = &DDStart[BE  *size_Mat];
				D.f[TW  ] = &DDStart[TW  *size_Mat];
				D.f[TN  ] = &DDStart[TN  *size_Mat];
				D.f[BS  ] = &DDStart[BS  *size_Mat];
				D.f[BN  ] = &DDStart[BN  *size_Mat];
				D.f[TS  ] = &DDStart[TS  *size_Mat];
				D.f[REST] = &DDStart[REST*size_Mat];
				D.f[TNE ] = &DDStart[TNE *size_Mat];
				D.f[TSW ] = &DDStart[TSW *size_Mat];
				D.f[TSE ] = &DDStart[TSE *size_Mat];
				D.f[TNW ] = &DDStart[TNW *size_Mat];
				D.f[BNE ] = &DDStart[BNE *size_Mat];
				D.f[BSW ] = &DDStart[BSW *size_Mat];
				D.f[BSE ] = &DDStart[BSE *size_Mat];
				D.f[BNW ] = &DDStart[BNW *size_Mat];
			}
			else
			{
				D.f[W   ] = &DDStart[E   *size_Mat];
				D.f[E   ] = &DDStart[W   *size_Mat];
				D.f[S   ] = &DDStart[N   *size_Mat];
				D.f[N   ] = &DDStart[S   *size_Mat];
				D.f[B   ] = &DDStart[T   *size_Mat];
				D.f[T   ] = &DDStart[B   *size_Mat];
				D.f[SW  ] = &DDStart[NE  *size_Mat];
				D.f[NE  ] = &DDStart[SW  *size_Mat];
				D.f[NW  ] = &DDStart[SE  *size_Mat];
				D.f[SE  ] = &DDStart[NW  *size_Mat];
				D.f[BW  ] = &DDStart[TE  *size_Mat];
				D.f[TE  ] = &DDStart[BW  *size_Mat];
				D.f[TW  ] = &DDStart[BE  *size_Mat];
				D.f[BE  ] = &DDStart[TW  *size_Mat];
				D.f[BS  ] = &DDStart[TN  *size_Mat];
				D.f[TN  ] = &DDStart[BS  *size_Mat];
				D.f[TS  ] = &DDStart[BN  *size_Mat];
				D.f[BN  ] = &DDStart[TS  *size_Mat];
				D.f[REST] = &DDStart[REST*size_Mat];
				D.f[BSW ] = &DDStart[TNE *size_Mat];
				D.f[BNE ] = &DDStart[TSW *size_Mat];
				D.f[BNW ] = &DDStart[TSE *size_Mat];
				D.f[BSE ] = &DDStart[TNW *size_Mat];
				D.f[TSW ] = &DDStart[BNE *size_Mat];
				D.f[TNE ] = &DDStart[BSW *size_Mat];
				D.f[TNW ] = &DDStart[BSE *size_Mat];
				D.f[TSE ] = &DDStart[BNW *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			unsigned int kw   = neighborX[k];
			unsigned int ks   = neighborY[k];
			unsigned int kb   = neighborZ[k];
			unsigned int ksw  = neighborY[kw];
			unsigned int kbw  = neighborZ[kw];
			unsigned int kbs  = neighborZ[ks];
			unsigned int kbsw = neighborZ[ksw];
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[E   ])[k  ];
			real mfabb = (D.f[W   ])[kw ];
			real mfbcb = (D.f[N   ])[k  ];
			real mfbab = (D.f[S   ])[ks ];
			real mfbbc = (D.f[T   ])[k  ];
			real mfbba = (D.f[B   ])[kb ];
			real mfccb = (D.f[NE  ])[k  ];
			real mfaab = (D.f[SW  ])[ksw];
			real mfcab = (D.f[SE  ])[ks ];
			real mfacb = (D.f[NW  ])[kw ];
			real mfcbc = (D.f[TE  ])[k  ];
			real mfaba = (D.f[BW  ])[kbw];
			real mfcba = (D.f[BE  ])[kb ];
			real mfabc = (D.f[TW  ])[kw ];
			real mfbcc = (D.f[TN  ])[k  ];
			real mfbaa = (D.f[BS  ])[kbs];
			real mfbca = (D.f[BN  ])[kb ];
			real mfbac = (D.f[TS  ])[ks ];
			real mfbbb = (D.f[REST])[k  ];
			real mfccc = (D.f[TNE ])[k  ];
			real mfaac = (D.f[TSW ])[ksw];
			real mfcac = (D.f[TSE ])[ks ];
			real mfacc = (D.f[TNW ])[kw ];
			real mfcca = (D.f[BNE ])[kb ];
			real mfaaa = (D.f[BSW ])[kbsw];
			real mfcaa = (D.f[BSE ])[kbs];
			real mfaca = (D.f[BNW ])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			real vvx    =((((mfccc-mfaaa) + (mfcac-mfaca)) + ((mfcaa-mfacc) + (mfcca-mfaac))) + 
						     (((mfcba-mfabc) + (mfcbc-mfaba)) + ((mfcab-mfacb) + (mfccb-mfaab))) +
						       (mfcbb-mfabb));
			real vvy    =((((mfccc-mfaaa) + (mfaca-mfcac)) + ((mfacc-mfcaa) + (mfcca-mfaac))) + 
				             (((mfbca-mfbac) + (mfbcc-mfbaa)) + ((mfacb-mfcab) + (mfccb-mfaab))) +
				               (mfbcb-mfbab));
			real vvz    =((((mfccc-mfaaa) + (mfcac-mfaca)) + ((mfacc-mfcaa) + (mfaac-mfcca))) + 
				             (((mfbac-mfbca) + (mfbcc-mfbaa)) + ((mfabc-mfcba) + (mfcbc-mfaba))) +
				               (mfbbc-mfbba));
			////////////////////////////////////////////////////////////////////////////////////
			real oMdrho = c1o1 - (mfccc+mfaaa + mfaca+mfcac + mfacc+mfcaa + mfaac+mfcca + 
								   mfbac+mfbca + mfbaa+mfbcc + mfabc+mfcba + mfaba+mfcbc + mfacb+mfcab + mfaab+mfccb +
								   mfabb+mfcbb + mfbab+mfbcb + mfbba+mfbbc + mfbbb);
			////////////////////////////////////////////////////////////////////////////////////
			real m0, m1, m2;	
			real vx2;
			real vy2;
			real vz2;
			vx2=vvx*vvx;
			vy2=vvy*vvy;
			vz2=vvz*vvz;
			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m2    = mfaaa	+ mfaac;
			m1    = mfaac	- mfaaa;
			m0    = m2		+ mfaab;
			mfaaa = m0;
			m0   += c1o36 * oMdrho;	
			mfaab = m1 -		m0 * vvz;
			mfaac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfabc;
			m1    = mfabc  - mfaba;
			m0    = m2		+ mfabb;
			mfaba = m0;
			m0   += c1o9 * oMdrho;
			mfabb = m1 -		m0 * vvz;
			mfabc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfacc;
			m1    = mfacc  - mfaca;
			m0    = m2		+ mfacb;
			mfaca = m0;
			m0   += c1o36 * oMdrho;
			mfacb = m1 -		m0 * vvz;
			mfacc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbac;
			m1    = mfbac	- mfbaa;
			m0    = m2		+ mfbab;
			mfbaa = m0;
			m0   += c1o9 * oMdrho;
			mfbab = m1 -		m0 * vvz;
			mfbac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbba  + mfbbc;
			m1    = mfbbc  - mfbba;
			m0    = m2		+ mfbbb;
			mfbba = m0;
			m0   += c4o9 * oMdrho;
			mfbbb = m1 -		m0 * vvz;
			mfbbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbca  + mfbcc;
			m1    = mfbcc  - mfbca;
			m0    = m2		+ mfbcb;
			mfbca = m0;
			m0   += c1o9 * oMdrho;
			mfbcb = m1 -		m0 * vvz;
			mfbcc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcac;
			m1    = mfcac	- mfcaa;
			m0    = m2		+ mfcab;
			mfcaa = m0;
			m0   += c1o36 * oMdrho;
			mfcab = m1 -		m0 * vvz;
			mfcac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcba  + mfcbc;
			m1    = mfcbc  - mfcba;
			m0    = m2		+ mfcbb;
			mfcba = m0;
			m0   += c1o9 * oMdrho;
			mfcbb = m1 -		m0 * vvz;
			mfcbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcca  + mfccc;
			m1    = mfccc  - mfcca;
			m0    = m2		+ mfccb;
			mfcca = m0;
			m0   += c1o36 * oMdrho;
			mfccb = m1 -		m0 * vvz;
			mfccc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2    = mfaaa	+ mfaca;
			m1    = mfaca	- mfaaa;
			m0    = m2		+ mfaba;
			mfaaa = m0;
			m0   += c1o6 * oMdrho;
			mfaba = m1 -		m0 * vvy;
			mfaca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab  + mfacb;
			m1    = mfacb  - mfaab;
			m0    = m2		+ mfabb;
			mfaab = m0;
			mfabb = m1 -		m0 * vvy;
			mfacb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac  + mfacc;
			m1    = mfacc  - mfaac;
			m0    = m2		+ mfabc;
			mfaac = m0;
			m0   += c1o18 * oMdrho;
			mfabc = m1 -		m0 * vvy;
			mfacc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbca;
			m1    = mfbca	- mfbaa;
			m0    = m2		+ mfbba;
			mfbaa = m0;
			m0   += c2o3 * oMdrho;
			mfbba = m1 -		m0 * vvy;
			mfbca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbab  + mfbcb;
			m1    = mfbcb  - mfbab;
			m0    = m2		+ mfbbb;
			mfbab = m0;
			mfbbb = m1 -		m0 * vvy;
			mfbcb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbac  + mfbcc;
			m1    = mfbcc  - mfbac;
			m0    = m2		+ mfbbc;
			mfbac = m0;
			m0   += c2o9 * oMdrho;
			mfbbc = m1 -		m0 * vvy;
			mfbcc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcca;
			m1    = mfcca	- mfcaa;
			m0    = m2		+ mfcba;
			mfcaa = m0;
			m0   += c1o6 * oMdrho;
			mfcba = m1 -		m0 * vvy;
			mfcca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcab  + mfccb;
			m1    = mfccb  - mfcab;
			m0    = m2		+ mfcbb;
			mfcab = m0;
			mfcbb = m1 -		m0 * vvy;
			mfccb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcac  + mfccc;
			m1    = mfccc  - mfcac;
			m0    = m2		+ mfcbc;
			mfcac = m0;
			m0   += c1o18 * oMdrho;
			mfcbc = m1 -		m0 * vvy;
			mfccc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2    = mfaaa	+ mfcaa;
			m1    = mfcaa	- mfaaa;
			m0    = m2		+ mfbaa;
			mfaaa = m0;
			m0   += c1o1* oMdrho;
			mfbaa = m1 -		m0 * vvx;
			mfcaa = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfcba;
			m1    = mfcba  - mfaba;
			m0    = m2		+ mfbba;
			mfaba = m0;
			mfbba = m1 -		m0 * vvx;
			mfcba = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfcca;
			m1    = mfcca  - mfaca;
			m0    = m2		+ mfbca;
			mfaca = m0;
			m0   += c1o3 * oMdrho;
			mfbca = m1 -		m0 * vvx;
			mfcca = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab	+ mfcab;
			m1    = mfcab	- mfaab;
			m0    = m2		+ mfbab;
			mfaab = m0;
			mfbab = m1 -		m0 * vvx;
			mfcab = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabb  + mfcbb;
			m1    = mfcbb  - mfabb;
			m0    = m2		+ mfbbb;
			mfabb = m0;
			mfbbb = m1 -		m0 * vvx;
			mfcbb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacb  + mfccb;
			m1    = mfccb  - mfacb;
			m0    = m2		+ mfbcb;
			mfacb = m0;
			mfbcb = m1 -		m0 * vvx;
			mfccb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac	+ mfcac;
			m1    = mfcac	- mfaac;
			m0    = m2		+ mfbac;
			mfaac = m0;
			m0   += c1o3 * oMdrho;
			mfbac = m1 -		m0 * vvx;
			mfcac = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabc  + mfcbc;
			m1    = mfcbc  - mfabc;
			m0    = m2		+ mfbbc;
			mfabc = m0;
			mfbbc = m1 -		m0 * vvx;
			mfcbc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacc  + mfccc;
			m1    = mfccc  - mfacc;
			m0    = m2		+ mfbcc;
			mfacc = m0;
			m0   += c1o9 * oMdrho;
			mfbcc = m1 -		m0 * vvx;
			mfccc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			//3.
			CUMbbb[k] = mfbbb;
			CUMabc[k] = mfabc;
			CUMbac[k] = mfbac;
			CUMbca[k] = mfbca;
			CUMcba[k] = mfcba;
			CUMacb[k] = mfacb;
			CUMcab[k] = mfcab;
			////////////////////////////////////////////////////////////////////////////////////
		}                                                                                                                    
	}
}
////////////////////////////////////////////////////////////////////////////////































////////////////////////////////////////////////////////////////////////////////
__global__ void LBCalc3rdMomentsCompSP27(real* CUMbbb,
													real* CUMabc,
													real* CUMbac,
													real* CUMbca,
													real* CUMcba,
													real* CUMacb,
													real* CUMcab,
													unsigned int* bcMatD,
													unsigned int* neighborX,
													unsigned int* neighborY,
													unsigned int* neighborZ,
													real* DDStart,
													int size_Mat,
													bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if(k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if( (BC != GEO_SOLID) && (BC != GEO_VOID) )
		{
			Distributions27 D;
			if (EvenOrOdd==true)
			{
				D.f[E   ] = &DDStart[E   *size_Mat];
				D.f[W   ] = &DDStart[W   *size_Mat];
				D.f[N   ] = &DDStart[N   *size_Mat];
				D.f[S   ] = &DDStart[S   *size_Mat];
				D.f[T   ] = &DDStart[T   *size_Mat];
				D.f[B   ] = &DDStart[B   *size_Mat];
				D.f[NE  ] = &DDStart[NE  *size_Mat];
				D.f[SW  ] = &DDStart[SW  *size_Mat];
				D.f[SE  ] = &DDStart[SE  *size_Mat];
				D.f[NW  ] = &DDStart[NW  *size_Mat];
				D.f[TE  ] = &DDStart[TE  *size_Mat];
				D.f[BW  ] = &DDStart[BW  *size_Mat];
				D.f[BE  ] = &DDStart[BE  *size_Mat];
				D.f[TW  ] = &DDStart[TW  *size_Mat];
				D.f[TN  ] = &DDStart[TN  *size_Mat];
				D.f[BS  ] = &DDStart[BS  *size_Mat];
				D.f[BN  ] = &DDStart[BN  *size_Mat];
				D.f[TS  ] = &DDStart[TS  *size_Mat];
				D.f[REST] = &DDStart[REST*size_Mat];
				D.f[TNE ] = &DDStart[TNE *size_Mat];
				D.f[TSW ] = &DDStart[TSW *size_Mat];
				D.f[TSE ] = &DDStart[TSE *size_Mat];
				D.f[TNW ] = &DDStart[TNW *size_Mat];
				D.f[BNE ] = &DDStart[BNE *size_Mat];
				D.f[BSW ] = &DDStart[BSW *size_Mat];
				D.f[BSE ] = &DDStart[BSE *size_Mat];
				D.f[BNW ] = &DDStart[BNW *size_Mat];
			}
			else
			{
				D.f[W   ] = &DDStart[E   *size_Mat];
				D.f[E   ] = &DDStart[W   *size_Mat];
				D.f[S   ] = &DDStart[N   *size_Mat];
				D.f[N   ] = &DDStart[S   *size_Mat];
				D.f[B   ] = &DDStart[T   *size_Mat];
				D.f[T   ] = &DDStart[B   *size_Mat];
				D.f[SW  ] = &DDStart[NE  *size_Mat];
				D.f[NE  ] = &DDStart[SW  *size_Mat];
				D.f[NW  ] = &DDStart[SE  *size_Mat];
				D.f[SE  ] = &DDStart[NW  *size_Mat];
				D.f[BW  ] = &DDStart[TE  *size_Mat];
				D.f[TE  ] = &DDStart[BW  *size_Mat];
				D.f[TW  ] = &DDStart[BE  *size_Mat];
				D.f[BE  ] = &DDStart[TW  *size_Mat];
				D.f[BS  ] = &DDStart[TN  *size_Mat];
				D.f[TN  ] = &DDStart[BS  *size_Mat];
				D.f[TS  ] = &DDStart[BN  *size_Mat];
				D.f[BN  ] = &DDStart[TS  *size_Mat];
				D.f[REST] = &DDStart[REST*size_Mat];
				D.f[BSW ] = &DDStart[TNE *size_Mat];
				D.f[BNE ] = &DDStart[TSW *size_Mat];
				D.f[BNW ] = &DDStart[TSE *size_Mat];
				D.f[BSE ] = &DDStart[TNW *size_Mat];
				D.f[TSW ] = &DDStart[BNE *size_Mat];
				D.f[TNE ] = &DDStart[BSW *size_Mat];
				D.f[TNW ] = &DDStart[BSE *size_Mat];
				D.f[TSE ] = &DDStart[BNW *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			unsigned int kw   = neighborX[k];
			unsigned int ks   = neighborY[k];
			unsigned int kb   = neighborZ[k];
			unsigned int ksw  = neighborY[kw];
			unsigned int kbw  = neighborZ[kw];
			unsigned int kbs  = neighborZ[ks];
			unsigned int kbsw = neighborZ[ksw];
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[E   ])[k  ];
			real mfabb = (D.f[W   ])[kw ];
			real mfbcb = (D.f[N   ])[k  ];
			real mfbab = (D.f[S   ])[ks ];
			real mfbbc = (D.f[T   ])[k  ];
			real mfbba = (D.f[B   ])[kb ];
			real mfccb = (D.f[NE  ])[k  ];
			real mfaab = (D.f[SW  ])[ksw];
			real mfcab = (D.f[SE  ])[ks ];
			real mfacb = (D.f[NW  ])[kw ];
			real mfcbc = (D.f[TE  ])[k  ];
			real mfaba = (D.f[BW  ])[kbw];
			real mfcba = (D.f[BE  ])[kb ];
			real mfabc = (D.f[TW  ])[kw ];
			real mfbcc = (D.f[TN  ])[k  ];
			real mfbaa = (D.f[BS  ])[kbs];
			real mfbca = (D.f[BN  ])[kb ];
			real mfbac = (D.f[TS  ])[ks ];
			real mfbbb = (D.f[REST])[k  ];
			real mfccc = (D.f[TNE ])[k  ];
			real mfaac = (D.f[TSW ])[ksw];
			real mfcac = (D.f[TSE ])[ks ];
			real mfacc = (D.f[TNW ])[kw ];
			real mfcca = (D.f[BNE ])[kb ];
			real mfaaa = (D.f[BSW ])[kbsw];
			real mfcaa = (D.f[BSE ])[kbs];
			real mfaca = (D.f[BNW ])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			real drho = ((((mfccc+mfaaa) + (mfaca+mfcac)) + ((mfacc+mfcaa) + (mfaac+mfcca))) + 
							(((mfbac+mfbca) + (mfbaa+mfbcc)) + ((mfabc+mfcba) + (mfaba+mfcbc)) + ((mfacb+mfcab) + (mfaab+mfccb))) +
							((mfabb+mfcbb) + (mfbab+mfbcb)) + (mfbba+mfbbc)) + mfbbb;

			real rho = c1o1+drho;
			////////////////////////////////////////////////////////////////////////////////////
			real vvx    =((((mfccc-mfaaa) + (mfcac-mfaca)) + ((mfcaa-mfacc) + (mfcca-mfaac))) + 
						     (((mfcba-mfabc) + (mfcbc-mfaba)) + ((mfcab-mfacb) + (mfccb-mfaab))) +
						       (mfcbb-mfabb)) / rho;
			real vvy    =((((mfccc-mfaaa) + (mfaca-mfcac)) + ((mfacc-mfcaa) + (mfcca-mfaac))) + 
				             (((mfbca-mfbac) + (mfbcc-mfbaa)) + ((mfacb-mfcab) + (mfccb-mfaab))) +
				               (mfbcb-mfbab)) / rho;
			real vvz    =((((mfccc-mfaaa) + (mfcac-mfaca)) + ((mfacc-mfcaa) + (mfaac-mfcca))) + 
				             (((mfbac-mfbca) + (mfbcc-mfbaa)) + ((mfabc-mfcba) + (mfcbc-mfaba))) +
				               (mfbbc-mfbba)) / rho;
			////////////////////////////////////////////////////////////////////////////////////
			real oMdrho = c1o1; // comp special
			////////////////////////////////////////////////////////////////////////////////////
			real m0, m1, m2;	
			real vx2;
			real vy2;
			real vz2;
			vx2=vvx*vvx;
			vy2=vvy*vvy;
			vz2=vvz*vvz;
			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m2    = mfaaa	+ mfaac;
			m1    = mfaac	- mfaaa;
			m0    = m2		+ mfaab;
			mfaaa = m0;
			m0   += c1o36 * oMdrho;	
			mfaab = m1 -		m0 * vvz;
			mfaac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfabc;
			m1    = mfabc  - mfaba;
			m0    = m2		+ mfabb;
			mfaba = m0;
			m0   += c1o9 * oMdrho;
			mfabb = m1 -		m0 * vvz;
			mfabc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfacc;
			m1    = mfacc  - mfaca;
			m0    = m2		+ mfacb;
			mfaca = m0;
			m0   += c1o36 * oMdrho;
			mfacb = m1 -		m0 * vvz;
			mfacc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbac;
			m1    = mfbac	- mfbaa;
			m0    = m2		+ mfbab;
			mfbaa = m0;
			m0   += c1o9 * oMdrho;
			mfbab = m1 -		m0 * vvz;
			mfbac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbba  + mfbbc;
			m1    = mfbbc  - mfbba;
			m0    = m2		+ mfbbb;
			mfbba = m0;
			m0   += c4o9 * oMdrho;
			mfbbb = m1 -		m0 * vvz;
			mfbbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbca  + mfbcc;
			m1    = mfbcc  - mfbca;
			m0    = m2		+ mfbcb;
			mfbca = m0;
			m0   += c1o9 * oMdrho;
			mfbcb = m1 -		m0 * vvz;
			mfbcc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcac;
			m1    = mfcac	- mfcaa;
			m0    = m2		+ mfcab;
			mfcaa = m0;
			m0   += c1o36 * oMdrho;
			mfcab = m1 -		m0 * vvz;
			mfcac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcba  + mfcbc;
			m1    = mfcbc  - mfcba;
			m0    = m2		+ mfcbb;
			mfcba = m0;
			m0   += c1o9 * oMdrho;
			mfcbb = m1 -		m0 * vvz;
			mfcbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcca  + mfccc;
			m1    = mfccc  - mfcca;
			m0    = m2		+ mfccb;
			mfcca = m0;
			m0   += c1o36 * oMdrho;
			mfccb = m1 -		m0 * vvz;
			mfccc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2    = mfaaa	+ mfaca;
			m1    = mfaca	- mfaaa;
			m0    = m2		+ mfaba;
			mfaaa = m0;
			m0   += c1o6 * oMdrho;
			mfaba = m1 -		m0 * vvy;
			mfaca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab  + mfacb;
			m1    = mfacb  - mfaab;
			m0    = m2		+ mfabb;
			mfaab = m0;
			mfabb = m1 -		m0 * vvy;
			mfacb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac  + mfacc;
			m1    = mfacc  - mfaac;
			m0    = m2		+ mfabc;
			mfaac = m0;
			m0   += c1o18 * oMdrho;
			mfabc = m1 -		m0 * vvy;
			mfacc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbca;
			m1    = mfbca	- mfbaa;
			m0    = m2		+ mfbba;
			mfbaa = m0;
			m0   += c2o3 * oMdrho;
			mfbba = m1 -		m0 * vvy;
			mfbca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbab  + mfbcb;
			m1    = mfbcb  - mfbab;
			m0    = m2		+ mfbbb;
			mfbab = m0;
			mfbbb = m1 -		m0 * vvy;
			mfbcb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbac  + mfbcc;
			m1    = mfbcc  - mfbac;
			m0    = m2		+ mfbbc;
			mfbac = m0;
			m0   += c2o9 * oMdrho;
			mfbbc = m1 -		m0 * vvy;
			mfbcc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcca;
			m1    = mfcca	- mfcaa;
			m0    = m2		+ mfcba;
			mfcaa = m0;
			m0   += c1o6 * oMdrho;
			mfcba = m1 -		m0 * vvy;
			mfcca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcab  + mfccb;
			m1    = mfccb  - mfcab;
			m0    = m2		+ mfcbb;
			mfcab = m0;
			mfcbb = m1 -		m0 * vvy;
			mfccb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcac  + mfccc;
			m1    = mfccc  - mfcac;
			m0    = m2		+ mfcbc;
			mfcac = m0;
			m0   += c1o18 * oMdrho;
			mfcbc = m1 -		m0 * vvy;
			mfccc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2    = mfaaa	+ mfcaa;
			m1    = mfcaa	- mfaaa;
			m0    = m2		+ mfbaa;
			mfaaa = m0;
			m0   += c1o1* oMdrho;
			mfbaa = m1 -		m0 * vvx;
			mfcaa = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfcba;
			m1    = mfcba  - mfaba;
			m0    = m2		+ mfbba;
			mfaba = m0;
			mfbba = m1 -		m0 * vvx;
			mfcba = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfcca;
			m1    = mfcca  - mfaca;
			m0    = m2		+ mfbca;
			mfaca = m0;
			m0   += c1o3 * oMdrho;
			mfbca = m1 -		m0 * vvx;
			mfcca = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab	+ mfcab;
			m1    = mfcab	- mfaab;
			m0    = m2		+ mfbab;
			mfaab = m0;
			mfbab = m1 -		m0 * vvx;
			mfcab = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabb  + mfcbb;
			m1    = mfcbb  - mfabb;
			m0    = m2		+ mfbbb;
			mfabb = m0;
			mfbbb = m1 -		m0 * vvx;
			mfcbb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacb  + mfccb;
			m1    = mfccb  - mfacb;
			m0    = m2		+ mfbcb;
			mfacb = m0;
			mfbcb = m1 -		m0 * vvx;
			mfccb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac	+ mfcac;
			m1    = mfcac	- mfaac;
			m0    = m2		+ mfbac;
			mfaac = m0;
			m0   += c1o3 * oMdrho;
			mfbac = m1 -		m0 * vvx;
			mfcac = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabc  + mfcbc;
			m1    = mfcbc  - mfabc;
			m0    = m2		+ mfbbc;
			mfabc = m0;
			mfbbc = m1 -		m0 * vvx;
			mfcbc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacc  + mfccc;
			m1    = mfccc  - mfacc;
			m0    = m2		+ mfbcc;
			mfacc = m0;
			m0   += c1o9 * oMdrho;
			mfbcc = m1 -		m0 * vvx;
			mfccc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			//3.
			CUMbbb[k] = mfbbb;
			CUMabc[k] = mfabc;
			CUMbac[k] = mfbac;
			CUMbca[k] = mfbca;
			CUMcba[k] = mfcba;
			CUMacb[k] = mfacb;
			CUMcab[k] = mfcab;
			////////////////////////////////////////////////////////////////////////////////////
		}                                                                                                                    
	}
}
////////////////////////////////////////////////////////////////////////////////































////////////////////////////////////////////////////////////////////////////////
__global__ void LBCalcHigherMomentsIncompSP27(   real* CUMcbb,
															real* CUMbcb,
															real* CUMbbc,
															real* CUMcca,
															real* CUMcac,
															real* CUMacc,
															real* CUMbcc,
															real* CUMcbc,
															real* CUMccb,
															real* CUMccc,
															unsigned int* bcMatD,
															unsigned int* neighborX,
															unsigned int* neighborY,
															unsigned int* neighborZ,
															real* DDStart,
															int size_Mat,
															bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if(k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if( (BC != GEO_SOLID) && (BC != GEO_VOID) )
		{
			Distributions27 D;
			if (EvenOrOdd==true)
			{
				D.f[E   ] = &DDStart[E   *size_Mat];
				D.f[W   ] = &DDStart[W   *size_Mat];
				D.f[N   ] = &DDStart[N   *size_Mat];
				D.f[S   ] = &DDStart[S   *size_Mat];
				D.f[T   ] = &DDStart[T   *size_Mat];
				D.f[B   ] = &DDStart[B   *size_Mat];
				D.f[NE  ] = &DDStart[NE  *size_Mat];
				D.f[SW  ] = &DDStart[SW  *size_Mat];
				D.f[SE  ] = &DDStart[SE  *size_Mat];
				D.f[NW  ] = &DDStart[NW  *size_Mat];
				D.f[TE  ] = &DDStart[TE  *size_Mat];
				D.f[BW  ] = &DDStart[BW  *size_Mat];
				D.f[BE  ] = &DDStart[BE  *size_Mat];
				D.f[TW  ] = &DDStart[TW  *size_Mat];
				D.f[TN  ] = &DDStart[TN  *size_Mat];
				D.f[BS  ] = &DDStart[BS  *size_Mat];
				D.f[BN  ] = &DDStart[BN  *size_Mat];
				D.f[TS  ] = &DDStart[TS  *size_Mat];
				D.f[REST] = &DDStart[REST*size_Mat];
				D.f[TNE ] = &DDStart[TNE *size_Mat];
				D.f[TSW ] = &DDStart[TSW *size_Mat];
				D.f[TSE ] = &DDStart[TSE *size_Mat];
				D.f[TNW ] = &DDStart[TNW *size_Mat];
				D.f[BNE ] = &DDStart[BNE *size_Mat];
				D.f[BSW ] = &DDStart[BSW *size_Mat];
				D.f[BSE ] = &DDStart[BSE *size_Mat];
				D.f[BNW ] = &DDStart[BNW *size_Mat];
			}
			else
			{
				D.f[W   ] = &DDStart[E   *size_Mat];
				D.f[E   ] = &DDStart[W   *size_Mat];
				D.f[S   ] = &DDStart[N   *size_Mat];
				D.f[N   ] = &DDStart[S   *size_Mat];
				D.f[B   ] = &DDStart[T   *size_Mat];
				D.f[T   ] = &DDStart[B   *size_Mat];
				D.f[SW  ] = &DDStart[NE  *size_Mat];
				D.f[NE  ] = &DDStart[SW  *size_Mat];
				D.f[NW  ] = &DDStart[SE  *size_Mat];
				D.f[SE  ] = &DDStart[NW  *size_Mat];
				D.f[BW  ] = &DDStart[TE  *size_Mat];
				D.f[TE  ] = &DDStart[BW  *size_Mat];
				D.f[TW  ] = &DDStart[BE  *size_Mat];
				D.f[BE  ] = &DDStart[TW  *size_Mat];
				D.f[BS  ] = &DDStart[TN  *size_Mat];
				D.f[TN  ] = &DDStart[BS  *size_Mat];
				D.f[TS  ] = &DDStart[BN  *size_Mat];
				D.f[BN  ] = &DDStart[TS  *size_Mat];
				D.f[REST] = &DDStart[REST*size_Mat];
				D.f[BSW ] = &DDStart[TNE *size_Mat];
				D.f[BNE ] = &DDStart[TSW *size_Mat];
				D.f[BNW ] = &DDStart[TSE *size_Mat];
				D.f[BSE ] = &DDStart[TNW *size_Mat];
				D.f[TSW ] = &DDStart[BNE *size_Mat];
				D.f[TNE ] = &DDStart[BSW *size_Mat];
				D.f[TNW ] = &DDStart[BSE *size_Mat];
				D.f[TSE ] = &DDStart[BNW *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			unsigned int kw   = neighborX[k];
			unsigned int ks   = neighborY[k];
			unsigned int kb   = neighborZ[k];
			unsigned int ksw  = neighborY[kw];
			unsigned int kbw  = neighborZ[kw];
			unsigned int kbs  = neighborZ[ks];
			unsigned int kbsw = neighborZ[ksw];
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[E   ])[k  ];
			real mfabb = (D.f[W   ])[kw ];
			real mfbcb = (D.f[N   ])[k  ];
			real mfbab = (D.f[S   ])[ks ];
			real mfbbc = (D.f[T   ])[k  ];
			real mfbba = (D.f[B   ])[kb ];
			real mfccb = (D.f[NE  ])[k  ];
			real mfaab = (D.f[SW  ])[ksw];
			real mfcab = (D.f[SE  ])[ks ];
			real mfacb = (D.f[NW  ])[kw ];
			real mfcbc = (D.f[TE  ])[k  ];
			real mfaba = (D.f[BW  ])[kbw];
			real mfcba = (D.f[BE  ])[kb ];
			real mfabc = (D.f[TW  ])[kw ];
			real mfbcc = (D.f[TN  ])[k  ];
			real mfbaa = (D.f[BS  ])[kbs];
			real mfbca = (D.f[BN  ])[kb ];
			real mfbac = (D.f[TS  ])[ks ];
			real mfbbb = (D.f[REST])[k  ];
			real mfccc = (D.f[TNE ])[k  ];
			real mfaac = (D.f[TSW ])[ksw];
			real mfcac = (D.f[TSE ])[ks ];
			real mfacc = (D.f[TNW ])[kw ];
			real mfcca = (D.f[BNE ])[kb ];
			real mfaaa = (D.f[BSW ])[kbsw];
			real mfcaa = (D.f[BSE ])[kbs];
			real mfaca = (D.f[BNW ])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			real vvx    =((((mfccc-mfaaa) + (mfcac-mfaca)) + ((mfcaa-mfacc) + (mfcca-mfaac))) + 
						     (((mfcba-mfabc) + (mfcbc-mfaba)) + ((mfcab-mfacb) + (mfccb-mfaab))) +
						       (mfcbb-mfabb));
			real vvy    =((((mfccc-mfaaa) + (mfaca-mfcac)) + ((mfacc-mfcaa) + (mfcca-mfaac))) + 
				             (((mfbca-mfbac) + (mfbcc-mfbaa)) + ((mfacb-mfcab) + (mfccb-mfaab))) +
				               (mfbcb-mfbab));
			real vvz    =((((mfccc-mfaaa) + (mfcac-mfaca)) + ((mfacc-mfcaa) + (mfaac-mfcca))) + 
				             (((mfbac-mfbca) + (mfbcc-mfbaa)) + ((mfabc-mfcba) + (mfcbc-mfaba))) +
				               (mfbbc-mfbba));
			////////////////////////////////////////////////////////////////////////////////////
			real oMdrho = c1o1 - (mfccc+mfaaa + mfaca+mfcac + mfacc+mfcaa + mfaac+mfcca + 
								   mfbac+mfbca + mfbaa+mfbcc + mfabc+mfcba + mfaba+mfcbc + mfacb+mfcab + mfaab+mfccb +
								   mfabb+mfcbb + mfbab+mfbcb + mfbba+mfbbc + mfbbb);
			////////////////////////////////////////////////////////////////////////////////////
			real m0, m1, m2;	
			real vx2;
			real vy2;
			real vz2;
			vx2=vvx*vvx;
			vy2=vvy*vvy;
			vz2=vvz*vvz;
			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m2    = mfaaa	+ mfaac;
			m1    = mfaac	- mfaaa;
			m0    = m2		+ mfaab;
			mfaaa = m0;
			m0   += c1o36 * oMdrho;	
			mfaab = m1 -		m0 * vvz;
			mfaac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfabc;
			m1    = mfabc  - mfaba;
			m0    = m2		+ mfabb;
			mfaba = m0;
			m0   += c1o9 * oMdrho;
			mfabb = m1 -		m0 * vvz;
			mfabc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfacc;
			m1    = mfacc  - mfaca;
			m0    = m2		+ mfacb;
			mfaca = m0;
			m0   += c1o36 * oMdrho;
			mfacb = m1 -		m0 * vvz;
			mfacc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbac;
			m1    = mfbac	- mfbaa;
			m0    = m2		+ mfbab;
			mfbaa = m0;
			m0   += c1o9 * oMdrho;
			mfbab = m1 -		m0 * vvz;
			mfbac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbba  + mfbbc;
			m1    = mfbbc  - mfbba;
			m0    = m2		+ mfbbb;
			mfbba = m0;
			m0   += c4o9 * oMdrho;
			mfbbb = m1 -		m0 * vvz;
			mfbbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbca  + mfbcc;
			m1    = mfbcc  - mfbca;
			m0    = m2		+ mfbcb;
			mfbca = m0;
			m0   += c1o9 * oMdrho;
			mfbcb = m1 -		m0 * vvz;
			mfbcc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcac;
			m1    = mfcac	- mfcaa;
			m0    = m2		+ mfcab;
			mfcaa = m0;
			m0   += c1o36 * oMdrho;
			mfcab = m1 -		m0 * vvz;
			mfcac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcba  + mfcbc;
			m1    = mfcbc  - mfcba;
			m0    = m2		+ mfcbb;
			mfcba = m0;
			m0   += c1o9 * oMdrho;
			mfcbb = m1 -		m0 * vvz;
			mfcbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcca  + mfccc;
			m1    = mfccc  - mfcca;
			m0    = m2		+ mfccb;
			mfcca = m0;
			m0   += c1o36 * oMdrho;
			mfccb = m1 -		m0 * vvz;
			mfccc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2    = mfaaa	+ mfaca;
			m1    = mfaca	- mfaaa;
			m0    = m2		+ mfaba;
			mfaaa = m0;
			m0   += c1o6 * oMdrho;
			mfaba = m1 -		m0 * vvy;
			mfaca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab  + mfacb;
			m1    = mfacb  - mfaab;
			m0    = m2		+ mfabb;
			mfaab = m0;
			mfabb = m1 -		m0 * vvy;
			mfacb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac  + mfacc;
			m1    = mfacc  - mfaac;
			m0    = m2		+ mfabc;
			mfaac = m0;
			m0   += c1o18 * oMdrho;
			mfabc = m1 -		m0 * vvy;
			mfacc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbca;
			m1    = mfbca	- mfbaa;
			m0    = m2		+ mfbba;
			mfbaa = m0;
			m0   += c2o3 * oMdrho;
			mfbba = m1 -		m0 * vvy;
			mfbca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbab  + mfbcb;
			m1    = mfbcb  - mfbab;
			m0    = m2		+ mfbbb;
			mfbab = m0;
			mfbbb = m1 -		m0 * vvy;
			mfbcb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbac  + mfbcc;
			m1    = mfbcc  - mfbac;
			m0    = m2		+ mfbbc;
			mfbac = m0;
			m0   += c2o9 * oMdrho;
			mfbbc = m1 -		m0 * vvy;
			mfbcc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcca;
			m1    = mfcca	- mfcaa;
			m0    = m2		+ mfcba;
			mfcaa = m0;
			m0   += c1o6 * oMdrho;
			mfcba = m1 -		m0 * vvy;
			mfcca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcab  + mfccb;
			m1    = mfccb  - mfcab;
			m0    = m2		+ mfcbb;
			mfcab = m0;
			mfcbb = m1 -		m0 * vvy;
			mfccb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcac  + mfccc;
			m1    = mfccc  - mfcac;
			m0    = m2		+ mfcbc;
			mfcac = m0;
			m0   += c1o18 * oMdrho;
			mfcbc = m1 -		m0 * vvy;
			mfccc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2    = mfaaa	+ mfcaa;
			m1    = mfcaa	- mfaaa;
			m0    = m2		+ mfbaa;
			mfaaa = m0;
			m0   += c1o1* oMdrho;
			mfbaa = m1 -		m0 * vvx;
			mfcaa = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfcba;
			m1    = mfcba  - mfaba;
			m0    = m2		+ mfbba;
			mfaba = m0;
			mfbba = m1 -		m0 * vvx;
			mfcba = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfcca;
			m1    = mfcca  - mfaca;
			m0    = m2		+ mfbca;
			mfaca = m0;
			m0   += c1o3 * oMdrho;
			mfbca = m1 -		m0 * vvx;
			mfcca = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab	+ mfcab;
			m1    = mfcab	- mfaab;
			m0    = m2		+ mfbab;
			mfaab = m0;
			mfbab = m1 -		m0 * vvx;
			mfcab = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabb  + mfcbb;
			m1    = mfcbb  - mfabb;
			m0    = m2		+ mfbbb;
			mfabb = m0;
			mfbbb = m1 -		m0 * vvx;
			mfcbb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacb  + mfccb;
			m1    = mfccb  - mfacb;
			m0    = m2		+ mfbcb;
			mfacb = m0;
			mfbcb = m1 -		m0 * vvx;
			mfccb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac	+ mfcac;
			m1    = mfcac	- mfaac;
			m0    = m2		+ mfbac;
			mfaac = m0;
			m0   += c1o3 * oMdrho;
			mfbac = m1 -		m0 * vvx;
			mfcac = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabc  + mfcbc;
			m1    = mfcbc  - mfabc;
			m0    = m2		+ mfbbc;
			mfabc = m0;
			mfbbc = m1 -		m0 * vvx;
			mfcbc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacc  + mfccc;
			m1    = mfccc  - mfacc;
			m0    = m2		+ mfbcc;
			mfacc = m0;
			m0   += c1o9 * oMdrho;
			mfbcc = m1 -		m0 * vvx;
			mfccc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			//Cum 4.
			CUMcbb[k]      = mfcbb - ((mfcaa + c1o3 * oMdrho) * mfabb + c2o1 * mfbba * mfbab); 
			CUMbcb[k]      = mfbcb - ((mfaca + c1o3 * oMdrho) * mfbab + c2o1 * mfbba * mfabb); 
			CUMbbc[k]      = mfbbc - ((mfaac + c1o3 * oMdrho) * mfbba + c2o1 * mfbab * mfabb); 

			CUMcca[k]      = mfcca - ((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca) * oMdrho + c1o9*(oMdrho-c1o1)*oMdrho);
			CUMcac[k]      = mfcac - ((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac) * oMdrho + c1o9*(oMdrho-c1o1)*oMdrho);
			CUMacc[k]      = mfacc - ((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca) * oMdrho + c1o9*(oMdrho-c1o1)*oMdrho);

			//Cum 5.
			CUMbcc[k]      = mfbcc - (mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) - c1o3 * (mfbca + mfbac) * oMdrho;
			CUMcbc[k]      = mfcbc - (mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) - c1o3 * (mfcba + mfabc) * oMdrho;
			CUMccb[k]      = mfccb - (mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) - c1o3 * (mfacb + mfcab) * oMdrho;

			//Cum 6.
			CUMccc[k]      = mfccc  +((-c4o1 *  mfbbb * mfbbb  
							-           (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
							-    c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
							-     c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb))
							+(   c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
							+     c2o1 * (mfcaa * mfaca * mfaac)
							+ c16o1 *  mfbba * mfbab * mfabb)
							-    c1o3 * (mfacc + mfcac + mfcca) * oMdrho  -c1o9*oMdrho*oMdrho
							-    c1o9 * (mfcaa + mfaca + mfaac) * oMdrho*(c1o1-c2o1* oMdrho)- c1o27* oMdrho * oMdrho*(-c2o1* oMdrho)
							+(    c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
							+           (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa)) * c2o3*oMdrho) +c1o27*oMdrho;
			////////////////////////////////////////////////////////////////////////////////////
		}                                                                                                                    
	}
}































////////////////////////////////////////////////////////////////////////////////
__global__ void LBCalcHigherMomentsCompSP27( real* CUMcbb,
														real* CUMbcb,
														real* CUMbbc,
														real* CUMcca,
														real* CUMcac,
														real* CUMacc,
														real* CUMbcc,
														real* CUMcbc,
														real* CUMccb,
														real* CUMccc,
														unsigned int* bcMatD,
														unsigned int* neighborX,
														unsigned int* neighborY,
														unsigned int* neighborZ,
														real* DDStart,
														int size_Mat,
														bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if(k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if( (BC != GEO_SOLID) && (BC != GEO_VOID) )
		{
			Distributions27 D;
			if (EvenOrOdd==true)
			{
				D.f[E   ] = &DDStart[E   *size_Mat];
				D.f[W   ] = &DDStart[W   *size_Mat];
				D.f[N   ] = &DDStart[N   *size_Mat];
				D.f[S   ] = &DDStart[S   *size_Mat];
				D.f[T   ] = &DDStart[T   *size_Mat];
				D.f[B   ] = &DDStart[B   *size_Mat];
				D.f[NE  ] = &DDStart[NE  *size_Mat];
				D.f[SW  ] = &DDStart[SW  *size_Mat];
				D.f[SE  ] = &DDStart[SE  *size_Mat];
				D.f[NW  ] = &DDStart[NW  *size_Mat];
				D.f[TE  ] = &DDStart[TE  *size_Mat];
				D.f[BW  ] = &DDStart[BW  *size_Mat];
				D.f[BE  ] = &DDStart[BE  *size_Mat];
				D.f[TW  ] = &DDStart[TW  *size_Mat];
				D.f[TN  ] = &DDStart[TN  *size_Mat];
				D.f[BS  ] = &DDStart[BS  *size_Mat];
				D.f[BN  ] = &DDStart[BN  *size_Mat];
				D.f[TS  ] = &DDStart[TS  *size_Mat];
				D.f[REST] = &DDStart[REST*size_Mat];
				D.f[TNE ] = &DDStart[TNE *size_Mat];
				D.f[TSW ] = &DDStart[TSW *size_Mat];
				D.f[TSE ] = &DDStart[TSE *size_Mat];
				D.f[TNW ] = &DDStart[TNW *size_Mat];
				D.f[BNE ] = &DDStart[BNE *size_Mat];
				D.f[BSW ] = &DDStart[BSW *size_Mat];
				D.f[BSE ] = &DDStart[BSE *size_Mat];
				D.f[BNW ] = &DDStart[BNW *size_Mat];
			}
			else
			{
				D.f[W   ] = &DDStart[E   *size_Mat];
				D.f[E   ] = &DDStart[W   *size_Mat];
				D.f[S   ] = &DDStart[N   *size_Mat];
				D.f[N   ] = &DDStart[S   *size_Mat];
				D.f[B   ] = &DDStart[T   *size_Mat];
				D.f[T   ] = &DDStart[B   *size_Mat];
				D.f[SW  ] = &DDStart[NE  *size_Mat];
				D.f[NE  ] = &DDStart[SW  *size_Mat];
				D.f[NW  ] = &DDStart[SE  *size_Mat];
				D.f[SE  ] = &DDStart[NW  *size_Mat];
				D.f[BW  ] = &DDStart[TE  *size_Mat];
				D.f[TE  ] = &DDStart[BW  *size_Mat];
				D.f[TW  ] = &DDStart[BE  *size_Mat];
				D.f[BE  ] = &DDStart[TW  *size_Mat];
				D.f[BS  ] = &DDStart[TN  *size_Mat];
				D.f[TN  ] = &DDStart[BS  *size_Mat];
				D.f[TS  ] = &DDStart[BN  *size_Mat];
				D.f[BN  ] = &DDStart[TS  *size_Mat];
				D.f[REST] = &DDStart[REST*size_Mat];
				D.f[BSW ] = &DDStart[TNE *size_Mat];
				D.f[BNE ] = &DDStart[TSW *size_Mat];
				D.f[BNW ] = &DDStart[TSE *size_Mat];
				D.f[BSE ] = &DDStart[TNW *size_Mat];
				D.f[TSW ] = &DDStart[BNE *size_Mat];
				D.f[TNE ] = &DDStart[BSW *size_Mat];
				D.f[TNW ] = &DDStart[BSE *size_Mat];
				D.f[TSE ] = &DDStart[BNW *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			unsigned int kw   = neighborX[k];
			unsigned int ks   = neighborY[k];
			unsigned int kb   = neighborZ[k];
			unsigned int ksw  = neighborY[kw];
			unsigned int kbw  = neighborZ[kw];
			unsigned int kbs  = neighborZ[ks];
			unsigned int kbsw = neighborZ[ksw];
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[E   ])[k  ];
			real mfabb = (D.f[W   ])[kw ];
			real mfbcb = (D.f[N   ])[k  ];
			real mfbab = (D.f[S   ])[ks ];
			real mfbbc = (D.f[T   ])[k  ];
			real mfbba = (D.f[B   ])[kb ];
			real mfccb = (D.f[NE  ])[k  ];
			real mfaab = (D.f[SW  ])[ksw];
			real mfcab = (D.f[SE  ])[ks ];
			real mfacb = (D.f[NW  ])[kw ];
			real mfcbc = (D.f[TE  ])[k  ];
			real mfaba = (D.f[BW  ])[kbw];
			real mfcba = (D.f[BE  ])[kb ];
			real mfabc = (D.f[TW  ])[kw ];
			real mfbcc = (D.f[TN  ])[k  ];
			real mfbaa = (D.f[BS  ])[kbs];
			real mfbca = (D.f[BN  ])[kb ];
			real mfbac = (D.f[TS  ])[ks ];
			real mfbbb = (D.f[REST])[k  ];
			real mfccc = (D.f[TNE ])[k  ];
			real mfaac = (D.f[TSW ])[ksw];
			real mfcac = (D.f[TSE ])[ks ];
			real mfacc = (D.f[TNW ])[kw ];
			real mfcca = (D.f[BNE ])[kb ];
			real mfaaa = (D.f[BSW ])[kbsw];
			real mfcaa = (D.f[BSE ])[kbs];
			real mfaca = (D.f[BNW ])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			real drho = ((((mfccc+mfaaa) + (mfaca+mfcac)) + ((mfacc+mfcaa) + (mfaac+mfcca))) + 
							(((mfbac+mfbca) + (mfbaa+mfbcc)) + ((mfabc+mfcba) + (mfaba+mfcbc)) + ((mfacb+mfcab) + (mfaab+mfccb))) +
							((mfabb+mfcbb) + (mfbab+mfbcb)) + (mfbba+mfbbc)) + mfbbb;

			real rho = c1o1+drho;
			////////////////////////////////////////////////////////////////////////////////////
			real vvx    =((((mfccc-mfaaa) + (mfcac-mfaca)) + ((mfcaa-mfacc) + (mfcca-mfaac))) + 
						     (((mfcba-mfabc) + (mfcbc-mfaba)) + ((mfcab-mfacb) + (mfccb-mfaab))) +
						       (mfcbb-mfabb)) / rho;
			real vvy    =((((mfccc-mfaaa) + (mfaca-mfcac)) + ((mfacc-mfcaa) + (mfcca-mfaac))) + 
				             (((mfbca-mfbac) + (mfbcc-mfbaa)) + ((mfacb-mfcab) + (mfccb-mfaab))) +
				               (mfbcb-mfbab)) / rho;
			real vvz    =((((mfccc-mfaaa) + (mfcac-mfaca)) + ((mfacc-mfcaa) + (mfaac-mfcca))) + 
				             (((mfbac-mfbca) + (mfbcc-mfbaa)) + ((mfabc-mfcba) + (mfcbc-mfaba))) +
				               (mfbbc-mfbba)) / rho;
			////////////////////////////////////////////////////////////////////////////////////
			real oMdrho = c1o1; // comp special
			////////////////////////////////////////////////////////////////////////////////////
			real m0, m1, m2;	
			real vx2;
			real vy2;
			real vz2;
			vx2=vvx*vvx;
			vy2=vvy*vvy;
			vz2=vvz*vvz;
			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m2    = mfaaa	+ mfaac;
			m1    = mfaac	- mfaaa;
			m0    = m2		+ mfaab;
			mfaaa = m0;
			m0   += c1o36 * oMdrho;	
			mfaab = m1 -		m0 * vvz;
			mfaac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfabc;
			m1    = mfabc  - mfaba;
			m0    = m2		+ mfabb;
			mfaba = m0;
			m0   += c1o9 * oMdrho;
			mfabb = m1 -		m0 * vvz;
			mfabc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfacc;
			m1    = mfacc  - mfaca;
			m0    = m2		+ mfacb;
			mfaca = m0;
			m0   += c1o36 * oMdrho;
			mfacb = m1 -		m0 * vvz;
			mfacc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbac;
			m1    = mfbac	- mfbaa;
			m0    = m2		+ mfbab;
			mfbaa = m0;
			m0   += c1o9 * oMdrho;
			mfbab = m1 -		m0 * vvz;
			mfbac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbba  + mfbbc;
			m1    = mfbbc  - mfbba;
			m0    = m2		+ mfbbb;
			mfbba = m0;
			m0   += c4o9 * oMdrho;
			mfbbb = m1 -		m0 * vvz;
			mfbbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbca  + mfbcc;
			m1    = mfbcc  - mfbca;
			m0    = m2		+ mfbcb;
			mfbca = m0;
			m0   += c1o9 * oMdrho;
			mfbcb = m1 -		m0 * vvz;
			mfbcc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcac;
			m1    = mfcac	- mfcaa;
			m0    = m2		+ mfcab;
			mfcaa = m0;
			m0   += c1o36 * oMdrho;
			mfcab = m1 -		m0 * vvz;
			mfcac = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcba  + mfcbc;
			m1    = mfcbc  - mfcba;
			m0    = m2		+ mfcbb;
			mfcba = m0;
			m0   += c1o9 * oMdrho;
			mfcbb = m1 -		m0 * vvz;
			mfcbc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcca  + mfccc;
			m1    = mfccc  - mfcca;
			m0    = m2		+ mfccb;
			mfcca = m0;
			m0   += c1o36 * oMdrho;
			mfccb = m1 -		m0 * vvz;
			mfccc = m2 - c2o1*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2    = mfaaa	+ mfaca;
			m1    = mfaca	- mfaaa;
			m0    = m2		+ mfaba;
			mfaaa = m0;
			m0   += c1o6 * oMdrho;
			mfaba = m1 -		m0 * vvy;
			mfaca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab  + mfacb;
			m1    = mfacb  - mfaab;
			m0    = m2		+ mfabb;
			mfaab = m0;
			mfabb = m1 -		m0 * vvy;
			mfacb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac  + mfacc;
			m1    = mfacc  - mfaac;
			m0    = m2		+ mfabc;
			mfaac = m0;
			m0   += c1o18 * oMdrho;
			mfabc = m1 -		m0 * vvy;
			mfacc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbca;
			m1    = mfbca	- mfbaa;
			m0    = m2		+ mfbba;
			mfbaa = m0;
			m0   += c2o3 * oMdrho;
			mfbba = m1 -		m0 * vvy;
			mfbca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbab  + mfbcb;
			m1    = mfbcb  - mfbab;
			m0    = m2		+ mfbbb;
			mfbab = m0;
			mfbbb = m1 -		m0 * vvy;
			mfbcb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbac  + mfbcc;
			m1    = mfbcc  - mfbac;
			m0    = m2		+ mfbbc;
			mfbac = m0;
			m0   += c2o9 * oMdrho;
			mfbbc = m1 -		m0 * vvy;
			mfbcc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcca;
			m1    = mfcca	- mfcaa;
			m0    = m2		+ mfcba;
			mfcaa = m0;
			m0   += c1o6 * oMdrho;
			mfcba = m1 -		m0 * vvy;
			mfcca = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcab  + mfccb;
			m1    = mfccb  - mfcab;
			m0    = m2		+ mfcbb;
			mfcab = m0;
			mfcbb = m1 -		m0 * vvy;
			mfccb = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcac  + mfccc;
			m1    = mfccc  - mfcac;
			m0    = m2		+ mfcbc;
			mfcac = m0;
			m0   += c1o18 * oMdrho;
			mfcbc = m1 -		m0 * vvy;
			mfccc = m2 - c2o1*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2    = mfaaa	+ mfcaa;
			m1    = mfcaa	- mfaaa;
			m0    = m2		+ mfbaa;
			mfaaa = m0;
			m0   += c1o1* oMdrho;
			mfbaa = m1 -		m0 * vvx;
			mfcaa = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfcba;
			m1    = mfcba  - mfaba;
			m0    = m2		+ mfbba;
			mfaba = m0;
			mfbba = m1 -		m0 * vvx;
			mfcba = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfcca;
			m1    = mfcca  - mfaca;
			m0    = m2		+ mfbca;
			mfaca = m0;
			m0   += c1o3 * oMdrho;
			mfbca = m1 -		m0 * vvx;
			mfcca = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab	+ mfcab;
			m1    = mfcab	- mfaab;
			m0    = m2		+ mfbab;
			mfaab = m0;
			mfbab = m1 -		m0 * vvx;
			mfcab = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabb  + mfcbb;
			m1    = mfcbb  - mfabb;
			m0    = m2		+ mfbbb;
			mfabb = m0;
			mfbbb = m1 -		m0 * vvx;
			mfcbb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacb  + mfccb;
			m1    = mfccb  - mfacb;
			m0    = m2		+ mfbcb;
			mfacb = m0;
			mfbcb = m1 -		m0 * vvx;
			mfccb = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac	+ mfcac;
			m1    = mfcac	- mfaac;
			m0    = m2		+ mfbac;
			mfaac = m0;
			m0   += c1o3 * oMdrho;
			mfbac = m1 -		m0 * vvx;
			mfcac = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabc  + mfcbc;
			m1    = mfcbc  - mfabc;
			m0    = m2		+ mfbbc;
			mfabc = m0;
			mfbbc = m1 -		m0 * vvx;
			mfcbc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacc  + mfccc;
			m1    = mfccc  - mfacc;
			m0    = m2		+ mfbcc;
			mfacc = m0;
			m0   += c1o9 * oMdrho;
			mfbcc = m1 -		m0 * vvx;
			mfccc = m2 - c2o1*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////

			real OxxPyyPzz = c1o1;
			real omega = c1o1 / (c3o1*0.001 + c1o2);
			real B = (c4o1 * omega * OxxPyyPzz * (c9o1 * omega - c16o1) - c4o1 * omega * omega - c2o1 * OxxPyyPzz * OxxPyyPzz * (c2o1 + c9o1 * omega * (omega - c2o1))) /
				(c3o1 * (omega - OxxPyyPzz) * (OxxPyyPzz * (c2o1 + c3o1 * omega) - c8o1 * omega));

			CUMbcc[k] = mfbcc - ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)*(c1o1 + rho*c6o1*B / (c2o1 + c3o1 * B))) / rho;
			CUMcbc[k] = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)*(c1o1 + rho*c6o1*B / (c2o1 + c3o1 * B))) / rho;
			CUMccb[k] = mfccb - ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)*(c1o1 + rho*c6o1*B / (c2o1 + c3o1 * B))) / rho;

			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			//central moments to cumulants
			//4.
			CUMcbb[k]      = mfcbb - ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) / rho;	
			CUMbcb[k]      = mfbcb - ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) / rho; 
			CUMbbc[k]      = mfbbc - ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) / rho; 
			 		
			CUMcca[k]      = mfcca - (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho  - c1o9*(drho/rho));
			CUMcac[k]      = mfcac - (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho  - c1o9*(drho/rho));
			CUMacc[k]      = mfacc - (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho  - c1o9*(drho/rho));

			//5.
			//CUMbcc[k]      = mfbcc - ((mfaac * mfbca + mfaca * mfbac + four * mfabb * mfbbb + two * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac) ) / rho ;
			//CUMcbc[k]      = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + four * mfbab * mfbbb + two * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc) ) / rho ;
			//CUMccb[k]      = mfccb - ((mfcaa * mfacb + mfaca * mfcab + four * mfbba * mfbbb + two * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab) ) / rho ;
			
			//6.
			CUMccc[k]      = mfccc + ((-c4o1 *  mfbbb * mfbbb  
							-           (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
							-    c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
							-     c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
							+(   c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
							+     c2o1 * (mfcaa * mfaca * mfaac)
							+ c16o1 *  mfbba * mfbab * mfabb) / (rho * rho)
							-    c1o3 * (mfacc + mfcac + mfcca) /rho 
							-    c1o9 * (mfcaa + mfaca + mfaac) /rho 
							+(    c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba) 
							+           (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3 
							+ c1o27*((drho * drho - drho)/(rho*rho)));
			////////////////////////////////////////////////////////////////////////////////////
		}                                                                                                                    
	}
}
