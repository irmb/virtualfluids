#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file PrecursorBCs27.cu
//! \ingroup GPU
//! \author Henry Korb, Henrik Asmuth
//======================================================================================
#include "LBM/LB.h"
#include <lbm/constants/NumericConstants.h>
#include <lbm/constants/D3Q27.h>
#include <lbm/MacroscopicQuantities.h>

#include "LBM/GPUHelperFunctions/KernelUtilities.h"

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;
using namespace vf::gpu;

__global__ void QPrecursorDeviceCompZeroPress(
    int* subgridDistanceIndices,
    int numberOfBCnodes,
    int numberOfPrecursorNodes,
    int sizeQ,
    real omega,
    real* distributions,
    real* subgridDistances,
    uint* neighborX,
    uint* neighborY,
    uint* neighborZ,
    uint* neighbors0PP,
    uint* neighbors0PM,
    uint* neighbors0MP,
    uint* neighbors0MM,
    real* weights0PP,
    real* weights0PM,
    real* weights0MP,
    real* weights0MM,
    real* vLast,
    real* vCurrent,
    real velocityX,
    real velocityY,
    real velocityZ,
    real timeRatio,
    real velocityRatio,
    unsigned long long numberOfLBnodes,
    bool isEvenTimestep)
{
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned nodeIndex = getNodeIndex();

    if(nodeIndex>=numberOfBCnodes) return;

    ////////////////////////////////////////////////////////////////////////////////
    // interpolation of velocity
    real vxLastInterpd, vyLastInterpd, vzLastInterpd;
    real vxNextInterpd, vyNextInterpd, vzNextInterpd;

    uint kNeighbor0PP = neighbors0PP[nodeIndex];
    real d0PP = weights0PP[nodeIndex];

    real* vxLast = vLast;
    real* vyLast = &vLast[numberOfPrecursorNodes];
    real* vzLast = &vLast[2*numberOfPrecursorNodes];

    real* vxCurrent = vCurrent;
    real* vyCurrent = &vCurrent[numberOfPrecursorNodes];
    real* vzCurrent = &vCurrent[2*numberOfPrecursorNodes];

    if(d0PP < 1e6)
    {
        uint kNeighbor0PM = neighbors0PM[nodeIndex];
        uint kNeighbor0MP = neighbors0MP[nodeIndex];
        uint kNeighbor0MM = neighbors0MM[nodeIndex];

        real d0PM = weights0PM[nodeIndex];
        real d0MP = weights0MP[nodeIndex];
        real d0MM = weights0MM[nodeIndex];

        real invWeightSum = 1.f/(d0PP+d0PM+d0MP+d0MM);

        vxLastInterpd = (vxLast[kNeighbor0PP]*d0PP + vxLast[kNeighbor0PM]*d0PM + vxLast[kNeighbor0MP]*d0MP + vxLast[kNeighbor0MM]*d0MM)*invWeightSum;
        vyLastInterpd = (vyLast[kNeighbor0PP]*d0PP + vyLast[kNeighbor0PM]*d0PM + vyLast[kNeighbor0MP]*d0MP + vyLast[kNeighbor0MM]*d0MM)*invWeightSum;
        vzLastInterpd = (vzLast[kNeighbor0PP]*d0PP + vzLast[kNeighbor0PM]*d0PM + vzLast[kNeighbor0MP]*d0MP + vzLast[kNeighbor0MM]*d0MM)*invWeightSum;

        vxNextInterpd = (vxCurrent[kNeighbor0PP]*d0PP + vxCurrent[kNeighbor0PM]*d0PM + vxCurrent[kNeighbor0MP]*d0MP + vxCurrent[kNeighbor0MM]*d0MM)*invWeightSum;
        vyNextInterpd = (vyCurrent[kNeighbor0PP]*d0PP + vyCurrent[kNeighbor0PM]*d0PM + vyCurrent[kNeighbor0MP]*d0MP + vyCurrent[kNeighbor0MM]*d0MM)*invWeightSum;
        vzNextInterpd = (vzCurrent[kNeighbor0PP]*d0PP + vzCurrent[kNeighbor0PM]*d0PM + vzCurrent[kNeighbor0MP]*d0MP + vzCurrent[kNeighbor0MM]*d0MM)*invWeightSum;
    }
    else
    {
        vxLastInterpd = vxLast[kNeighbor0PP];
        vyLastInterpd = vyLast[kNeighbor0PP];
        vzLastInterpd = vzLast[kNeighbor0PP];

        vxNextInterpd = vxCurrent[kNeighbor0PP];
        vyNextInterpd = vyCurrent[kNeighbor0PP];
        vzNextInterpd = vzCurrent[kNeighbor0PP];
    }

    // if(k==16300)s printf("%f %f %f\n", vxLastInterpd, vyLastInterpd, vzLastInterpd);
    real VeloX = (velocityX + (1.f-timeRatio)*vxLastInterpd + timeRatio*vxNextInterpd)/velocityRatio;
    real VeloY = (velocityY + (1.f-timeRatio)*vyLastInterpd + timeRatio*vyNextInterpd)/velocityRatio;
    real VeloZ = (velocityZ + (1.f-timeRatio)*vzLastInterpd + timeRatio*vzNextInterpd)/velocityRatio;
    // From here on just a copy of QVelDeviceCompZeroPress
    ////////////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////////////////
    //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep
    //! is based on the esoteric twist algorithm \ref <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier
    //! et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
    //!
    Distributions27 dist;
    getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);

    unsigned int KQK  = subgridDistanceIndices[nodeIndex];
    unsigned int k000= KQK;
    unsigned int kP00   = KQK;
    unsigned int kM00   = neighborX[KQK];
    unsigned int k0P0   = KQK;
    unsigned int k0M0   = neighborY[KQK];
    unsigned int k00P   = KQK;
    unsigned int k00M   = neighborZ[KQK];
    unsigned int kMM0  = neighborY[kM00];
    unsigned int kPP0  = KQK;
    unsigned int kPM0  = k0M0;
    unsigned int kMP0  = kM00;
    unsigned int kM0M  = neighborZ[kM00];
    unsigned int kP0P  = KQK;
    unsigned int kP0M  = k00M;
    unsigned int kM0P  = kM00;
    unsigned int k0PP  = KQK;
    unsigned int k0MM  = neighborZ[k0M0];
    unsigned int k0PM  = k00M;
    unsigned int k0MP  = k0M0;
    unsigned int kPMP = k0M0;
    unsigned int kMPM = kM0M;
    unsigned int kMPP = kM00;
    unsigned int kPMM = k0MM;
    unsigned int kMMP = kMM0;
    unsigned int kPPM = k00M;
    unsigned int kPPP = KQK;
    unsigned int kMMM = neighborZ[kMM0];

    ////////////////////////////////////////////////////////////////////////////////
    //! - Set local distributions
    //!
    real f_M00 = (dist.f[DIR_P00])[kP00];
    real f_P00 = (dist.f[DIR_M00])[kM00];
    real f_0M0 = (dist.f[DIR_0P0])[k0P0];
    real f_0P0 = (dist.f[DIR_0M0])[k0M0];
    real f_00M = (dist.f[DIR_00P])[k00P];
    real f_00P = (dist.f[DIR_00M])[k00M];
    real f_MM0 = (dist.f[DIR_PP0])[kPP0];
    real f_PP0 = (dist.f[DIR_MM0])[kMM0];
    real f_MP0 = (dist.f[DIR_PM0])[kPM0];
    real f_PM0 = (dist.f[DIR_MP0])[kMP0];
    real f_M0M = (dist.f[DIR_P0P])[kP0P];
    real f_P0P = (dist.f[DIR_M0M])[kM0M];
    real f_M0P = (dist.f[DIR_P0M])[kP0M];
    real f_P0M = (dist.f[DIR_M0P])[kM0P];
    real f_0MM = (dist.f[DIR_0PP])[k0PP];
    real f_0PP = (dist.f[DIR_0MM])[k0MM];
    real f_0MP = (dist.f[DIR_0PM])[k0PM];
    real f_0PM = (dist.f[DIR_0MP])[k0MP];
    real f_MMM = (dist.f[DIR_PPP])[kPPP];
    real f_PPM = (dist.f[DIR_MMP])[kMMP];
    real f_MPM = (dist.f[DIR_PMP])[kPMP];
    real f_PMM = (dist.f[DIR_MPP])[kMPP];
    real f_MMP = (dist.f[DIR_PPM])[kPPM];
    real f_PPP = (dist.f[DIR_MMM])[kMMM];
    real f_MPP = (dist.f[DIR_PMM])[kPMM];
    real f_PMP = (dist.f[DIR_MPM])[kMPM];

    SubgridDistances27 subgridD;
    getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);

    ////////////////////////////////////////////////////////////////////////////////
      real drho   =  f_PMP + f_MPP + f_PPP + f_MMP + f_PMM + f_MPM + f_PPM + f_MMM +
                     f_0PM + f_0PP + f_0MP + f_0MM + f_P0M + f_M0P + f_P0P + f_M0M + f_PM0 + f_MP0 + f_PP0 + f_MM0 +
                     f_00P + f_00M + f_0P0 + f_0M0 + f_P00 + f_M00 + ((dist.f[DIR_000])[k000]);

      real vx1 =  (((f_PMP - f_MPM) - (f_MPP - f_PMM)) + ((f_PPP - f_MMM) - (f_MMP - f_PPM)) +
                      ((f_P0M - f_M0P)   + (f_P0P - f_M0M))   + ((f_PM0 - f_MP0)   + (f_PP0 - f_MM0)) +
                      (f_P00 - f_M00)) / (c1o1 + drho);


      real vx2 =   ((-(f_PMP - f_MPM) + (f_MPP - f_PMM)) + ((f_PPP - f_MMM) - (f_MMP - f_PPM)) +
                       ((f_0PM - f_0MP)   + (f_0PP - f_0MM))    + (-(f_PM0 - f_MP0)  + (f_PP0 - f_MM0)) +
                       (f_0P0 - f_0M0)) / (c1o1 + drho);

      real vx3 =   (((f_PMP - f_MPM) + (f_MPP - f_PMM)) + ((f_PPP - f_MMM) + (f_MMP - f_PPM)) +
                       (-(f_0PM - f_0MP)  + (f_0PP - f_0MM))   + ((f_P0P - f_M0M)   - (f_P0M - f_M0P)) +
                       (f_00P - f_00M)) / (c1o1 + drho);


    // if(k==16383 || k==0) printf("k %d kQ %d drho = %f u %f v %f w %f\n",k, KQK, drho, vx1, vx2, vx3);
      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);
    //////////////////////////////////////////////////////////////////////////


    ////////////////////////////////////////////////////////////////////////////////
    //! - Update distributions with subgrid distance (q) between zero and one
    real feq, q, velocityLB, velocityBC;
    q = (subgridD.q[DIR_P00])[nodeIndex];
    if (q>=c0o1 && q<=c1o1) // only update distribution for q between zero and one
    {
        velocityLB = vx1;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
        velocityBC = VeloX;
        (dist.f[DIR_M00])[kM00] = getInterpolatedDistributionForVeloWithPressureBC(q, f_P00, f_M00, feq, omega, drho, velocityBC, c2o27);
    }

    q = (subgridD.q[DIR_M00])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
        velocityBC = -VeloX;
        (dist.f[DIR_P00])[kP00] = getInterpolatedDistributionForVeloWithPressureBC(q, f_M00, f_P00, feq, omega, drho, velocityBC, c2o27);
    }

    q = (subgridD.q[DIR_0P0])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx2;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
        velocityBC = VeloY;
        (dist.f[DIR_0M0])[DIR_0M0] = getInterpolatedDistributionForVeloWithPressureBC(q, f_0P0, f_0M0, feq, omega, drho, velocityBC, c2o27);
    }

    q = (subgridD.q[DIR_0M0])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx2;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
        velocityBC = -VeloY;
        (dist.f[DIR_0P0])[k0P0] = getInterpolatedDistributionForVeloWithPressureBC(q, f_0M0, f_0P0, feq, omega, drho, velocityBC, c2o27);
    }

    q = (subgridD.q[DIR_00P])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
        velocityBC = VeloZ;
        (dist.f[DIR_00M])[k00M] = getInterpolatedDistributionForVeloWithPressureBC(q, f_00P, f_00M, feq, omega, drho, velocityBC, c2o27);
    }

    q = (subgridD.q[DIR_00M])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
        velocityBC = -VeloZ;
        (dist.f[DIR_00P])[k00P] = getInterpolatedDistributionForVeloWithPressureBC(q, f_00M, f_00P, feq, omega, drho, velocityBC, c2o27);
    }

    q = (subgridD.q[DIR_PP0])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx1 + vx2;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = VeloX + VeloY;
        (dist.f[DIR_MM0])[kMM0] = getInterpolatedDistributionForVeloWithPressureBC(q, f_PP0, f_MM0, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_MM0])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1 - vx2;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloX - VeloY;
        (dist.f[DIR_PP0])[kPP0] = getInterpolatedDistributionForVeloWithPressureBC(q, f_MM0, f_PP0, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_PM0])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx1 - vx2;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = VeloX - VeloY;
        (dist.f[DIR_MP0])[kMP0] = getInterpolatedDistributionForVeloWithPressureBC(q, f_PM0, f_MP0, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_MP0])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1 + vx2;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloX + VeloY;
        (dist.f[DIR_PM0])[kPM0] = getInterpolatedDistributionForVeloWithPressureBC(q, f_MP0, f_PM0, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_P0P])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx1 + vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = VeloX + VeloZ;
        (dist.f[DIR_M0M])[kM0M] = getInterpolatedDistributionForVeloWithPressureBC(q, f_P0P, f_M0M, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_M0M])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloX - VeloZ;
        (dist.f[DIR_P0P])[kP0P] = getInterpolatedDistributionForVeloWithPressureBC(q, f_M0M, f_P0P, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_P0M])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx1 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = VeloX - VeloZ;
        (dist.f[DIR_M0P])[kM0P] = getInterpolatedDistributionForVeloWithPressureBC(q, f_P0M, f_M0P, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_M0P])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1 + vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloX + VeloZ;
        (dist.f[DIR_P0M])[kP0M] = getInterpolatedDistributionForVeloWithPressureBC(q, f_M0P, f_P0M, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_0PP])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx2 + vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = VeloY + VeloZ;
        (dist.f[DIR_0MM])[k0MM] = getInterpolatedDistributionForVeloWithPressureBC(q, f_0PP, f_0MM, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_0MM])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx2 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloY - VeloZ;
        (dist.f[DIR_0PP])[k0PP] = getInterpolatedDistributionForVeloWithPressureBC(q, f_0MM, f_0PP, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_0PM])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx2 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = VeloY - VeloZ;
        (dist.f[DIR_0MP])[k0MP] = getInterpolatedDistributionForVeloWithPressureBC(q, f_0PM, f_0PP, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_0MP])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx2 + vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloY + VeloZ;
        (dist.f[DIR_0PM])[k0PM] = getInterpolatedDistributionForVeloWithPressureBC(q, f_0PP, f_0PM, feq, omega, drho, velocityBC, c1o54);
    }

    q = (subgridD.q[DIR_PPP])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx1 + vx2 + vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
        velocityBC = VeloX + VeloY + VeloZ;
        (dist.f[DIR_MMM])[kMMM] = getInterpolatedDistributionForVeloWithPressureBC(q, f_PPP, f_MMM, feq, omega, drho, velocityBC, c1o216);
    }

    q = (subgridD.q[DIR_MMM])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1 - vx2 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
        velocityBC = -VeloX - VeloY - VeloZ;
        (dist.f[DIR_PPP])[kPPP] = getInterpolatedDistributionForVeloWithPressureBC(q, f_MMM, f_PPP, feq, omega, drho, velocityBC, c1o216);
    }

    q = (subgridD.q[DIR_PPM])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx1 + vx2 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
        velocityBC = VeloX + VeloY - VeloZ;
        (dist.f[DIR_MMP])[kMMP] = getInterpolatedDistributionForVeloWithPressureBC(q, f_PPM, f_MMP, feq, omega, drho, velocityBC, c1o216);
    }

    q = (subgridD.q[DIR_MMP])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1 - vx2 + vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
        velocityBC = -VeloX - VeloY + VeloZ;
        (dist.f[DIR_PPM])[kPPM] = getInterpolatedDistributionForVeloWithPressureBC(q, f_MMP, f_PPM, feq, omega, drho, velocityBC, c1o216);
    }

    q = (subgridD.q[DIR_PMP])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx1 - vx2 + vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
        velocityBC = VeloX - VeloY + VeloZ;
        (dist.f[DIR_MPM])[kMPM] = getInterpolatedDistributionForVeloWithPressureBC(q, f_PMP, f_MPM, feq, omega, drho, velocityBC, c1o216);
    }

    q = (subgridD.q[DIR_MPM])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1 + vx2 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
        velocityBC = -VeloX + VeloY - VeloZ;
        (dist.f[DIR_PMP])[kPMP] = getInterpolatedDistributionForVeloWithPressureBC(q, f_MPM, f_PMP, feq, omega, drho, velocityBC, c1o216);
    }

    q = (subgridD.q[DIR_PMM])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = vx1 - vx2 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
        velocityBC = VeloX - VeloY - VeloZ;
        (dist.f[DIR_MPP])[kMPP] = getInterpolatedDistributionForVeloWithPressureBC(q, f_PMM, f_MPP, feq, omega, drho, velocityBC, c1o216);
    }

    q = (subgridD.q[DIR_MPP])[nodeIndex];
    if (q>=c0o1 && q<=c1o1)
    {
        velocityLB = -vx1 + vx2 + vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
        velocityBC = -VeloX + VeloY + VeloZ;
        (dist.f[DIR_PMM])[kPMM] = getInterpolatedDistributionForVeloWithPressureBC(q, f_MPP, f_PMM, feq, omega, drho, velocityBC, c1o216);
    }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////











































////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void PrecursorDeviceEQ27(
    int *subgridDistanceIndices,
    int numberOfBCnodes,
    int numberOfPrecursorNodes,
    real omega,
    real* distributions,
    uint* neighborX,
    uint* neighborY,
    uint* neighborZ,
    uint* neighbors0PP,
    uint* neighbors0PM,
    uint* neighbors0MP,
    uint* neighbors0MM,
    real* weights0PP,
    real* weights0PM,
    real* weights0MP,
    real* weights0MM,
    real* vLast,
    real* vCurrent,
    real velocityX,
    real velocityY,
    real velocityZ,
    real timeRatio,
    real velocityRatio,
    unsigned long long numberOfLBnodes,
    bool isEvenTimestep)
{
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned nodeIndex = getNodeIndex();

    if(nodeIndex>=numberOfBCnodes) return;

    ////////////////////////////////////////////////////////////////////////////////
    // interpolation of velocity
    real vxLastInterpd, vyLastInterpd, vzLastInterpd;
    real vxNextInterpd, vyNextInterpd, vzNextInterpd;

    uint kNeighbor0PP = neighbors0PP[nodeIndex];
    real d0PP = weights0PP[nodeIndex];

    real* vxLast = vLast;
    real* vyLast = &vLast[numberOfPrecursorNodes];
    real* vzLast = &vLast[2*numberOfPrecursorNodes];

    real* vxCurrent = vCurrent;
    real* vyCurrent = &vCurrent[numberOfPrecursorNodes];
    real* vzCurrent = &vCurrent[2*numberOfPrecursorNodes];

    if(d0PP < 1e6)
    {
        uint kNeighbor0PM = neighbors0PM[nodeIndex];
        uint kNeighbor0MP = neighbors0MP[nodeIndex];
        uint kNeighbor0MM = neighbors0MM[nodeIndex];

        real d0PM = weights0PM[nodeIndex];
        real d0MP = weights0MP[nodeIndex];
        real d0MM = weights0MM[nodeIndex];

        real invWeightSum = 1.f/(d0PP+d0PM+d0MP+d0MM);

        vxLastInterpd = (vxLast[kNeighbor0PP]*d0PP + vxLast[kNeighbor0PM]*d0PM + vxLast[kNeighbor0MP]*d0MP + vxLast[kNeighbor0MM]*d0MM)*invWeightSum;
        vyLastInterpd = (vyLast[kNeighbor0PP]*d0PP + vyLast[kNeighbor0PM]*d0PM + vyLast[kNeighbor0MP]*d0MP + vyLast[kNeighbor0MM]*d0MM)*invWeightSum;
        vzLastInterpd = (vzLast[kNeighbor0PP]*d0PP + vzLast[kNeighbor0PM]*d0PM + vzLast[kNeighbor0MP]*d0MP + vzLast[kNeighbor0MM]*d0MM)*invWeightSum;

        vxNextInterpd = (vxCurrent[kNeighbor0PP]*d0PP + vxCurrent[kNeighbor0PM]*d0PM + vxCurrent[kNeighbor0MP]*d0MP + vxCurrent[kNeighbor0MM]*d0MM)*invWeightSum;
        vyNextInterpd = (vyCurrent[kNeighbor0PP]*d0PP + vyCurrent[kNeighbor0PM]*d0PM + vyCurrent[kNeighbor0MP]*d0MP + vyCurrent[kNeighbor0MM]*d0MM)*invWeightSum;
        vzNextInterpd = (vzCurrent[kNeighbor0PP]*d0PP + vzCurrent[kNeighbor0PM]*d0PM + vzCurrent[kNeighbor0MP]*d0MP + vzCurrent[kNeighbor0MM]*d0MM)*invWeightSum;
    }
    else
    {
        vxLastInterpd = vxLast[kNeighbor0PP];
        vyLastInterpd = vyLast[kNeighbor0PP];
        vzLastInterpd = vzLast[kNeighbor0PP];

        vxNextInterpd = vxCurrent[kNeighbor0PP];
        vyNextInterpd = vyCurrent[kNeighbor0PP];
        vzNextInterpd = vzCurrent[kNeighbor0PP];
    }

    // if(k==16300) printf("%f %f %f\n", vxLastInterpd, vyLastInterpd, vzLastInterpd);
    real VeloX = (velocityX + (1.f-timeRatio)*vxLastInterpd + timeRatio*vxNextInterpd)/velocityRatio;
    real VeloY = (velocityY + (1.f-timeRatio)*vyLastInterpd + timeRatio*vyNextInterpd)/velocityRatio;
    real VeloZ = (velocityZ + (1.f-timeRatio)*vzLastInterpd + timeRatio*vzNextInterpd)/velocityRatio;
    // From here on just a copy of QVelDeviceCompZeroPress
    ////////////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////////////////
    //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep
    //! is based on the esoteric twist algorithm \ref <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier
    //! et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
    //!
    Distributions27 dist;
    getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

    unsigned int KQK  = subgridDistanceIndices[nodeIndex]; //QK
    unsigned int k000 = KQK; //000
    unsigned int kP00 = KQK; //P00
    unsigned int kM00 = neighborX[KQK]; //M00
    unsigned int k0P0   = KQK; //n
    unsigned int k0M0   = neighborY[KQK]; //s
    unsigned int k00P   = KQK; //t
    unsigned int k00M   = neighborZ[KQK]; //b
    unsigned int kMM0  = neighborY[kM00]; //sw
    unsigned int kPP0  = KQK; //ne
    unsigned int kPM0  = k0M0; //se
    unsigned int kMP0  = kM00; //nw
    unsigned int kM0M  = neighborZ[kM00]; //bw
    unsigned int kP0P  = KQK; //te
    unsigned int kP0M  = k00M; //be
    unsigned int k0PP  = KQK; //tn
    unsigned int k0MM  = neighborZ[k0M0]; //bs
    unsigned int kM0P  = kM00; //tw
    unsigned int k0PM  = k00M; //bn
    unsigned int k0MP  = k0M0; //ts
    unsigned int kPMP = k0M0; //tse
    unsigned int kMPM = kM0M; //bnw
    unsigned int kMPP = kM00; //tnw
    unsigned int kPMM = k0MM; //bse
    unsigned int kMMP = kMM0; //tsw
    unsigned int kPPM = k00M; //bne
    unsigned int kPPP = KQK; //tne
    unsigned int kMMM = neighborZ[kMM0]; //bsw

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // based on BGK Plus Comp
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    real f_M00 = (dist.f[DIR_P00])[kP00];
    real f_P00 = (dist.f[DIR_M00])[kM00];
    real f_0M0 = (dist.f[DIR_0P0])[k0P0];
    real f_0P0 = (dist.f[DIR_0M0])[k0M0];
    real f_00M = (dist.f[DIR_00P])[k00P];
    real f_00P = (dist.f[DIR_00M])[k00M];
    real f_MM0 = (dist.f[DIR_PP0])[kPP0];
    real f_PP0 = (dist.f[DIR_MM0])[kMM0];
    real f_MP0 = (dist.f[DIR_PM0])[kPM0];
    real f_PM0 = (dist.f[DIR_MP0])[kMP0];
    real f_M0M = (dist.f[DIR_P0P])[kP0P];
    real f_P0P = (dist.f[DIR_M0M])[kM0M];
    real f_M0P = (dist.f[DIR_P0M])[kP0M];
    real f_P0M = (dist.f[DIR_M0P])[kM0P];
    real f_0MM = (dist.f[DIR_0PP])[k0PP];
    real f_0PP = (dist.f[DIR_0MM])[k0MM];
    real f_0PM = (dist.f[DIR_0MP])[k0MP];
    real f_0MP = (dist.f[DIR_0PM])[k0PM];
    real f_000 = (dist.f[DIR_000])[k000];
    real f_MMM = (dist.f[DIR_PPP])[kPPP];
    real f_PPM = (dist.f[DIR_MMP])[kMMP];
    real f_MPM = (dist.f[DIR_PMP])[kPMP];
    real f_PMM = (dist.f[DIR_MPP])[kMPP];
    real f_MMP = (dist.f[DIR_PPM])[kPPM];
    real f_PPP = (dist.f[DIR_MMM])[kMMM];
    real f_MPP = (dist.f[DIR_PMM])[kPMM];
    real f_PMP = (dist.f[DIR_MPM])[kMPM];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set macroscopic quantities
      //!
      real drho = c0o1;

      real vx1  = VeloX;

      real vx2  = VeloY;

      real vx3  = VeloZ;

      real cusq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3);

      ////////////////////////////////////////////////////////////////////////////////
      f_000 = c8o27* (drho-(drho+c1o1)*cusq);
      f_P00 = c2o27* (drho+(drho+c1o1)*(c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cusq));
      f_M00 = c2o27* (drho+(drho+c1o1)*(c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cusq));
      f_0P0 = c2o27* (drho+(drho+c1o1)*(c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cusq));
      f_0M0 = c2o27* (drho+(drho+c1o1)*(c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cusq));
      f_00P = c2o27* (drho+(drho+c1o1)*(c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cusq));
      f_00M = c2o27* (drho+(drho+c1o1)*(c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cusq));
      f_PP0 = c1o54* (drho+(drho+c1o1)*(c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cusq));
      f_MM0 = c1o54* (drho+(drho+c1o1)*(c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cusq));
      f_PM0 = c1o54* (drho+(drho+c1o1)*(c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cusq));
      f_MP0 = c1o54* (drho+(drho+c1o1)*(c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cusq));
      f_P0P = c1o54* (drho+(drho+c1o1)*(c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cusq));
      f_M0M = c1o54* (drho+(drho+c1o1)*(c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cusq));
      f_P0M = c1o54* (drho+(drho+c1o1)*(c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cusq));
      f_M0P = c1o54* (drho+(drho+c1o1)*(c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cusq));
      f_0PP = c1o54* (drho+(drho+c1o1)*(c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cusq));
      f_0MM = c1o54* (drho+(drho+c1o1)*(c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cusq));
      f_0PM = c1o54* (drho+(drho+c1o1)*(c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cusq));
      f_0MP = c1o54* (drho+(drho+c1o1)*(c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cusq));
      f_PPP = c1o216*(drho+(drho+c1o1)*(c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cusq));
      f_MMM = c1o216*(drho+(drho+c1o1)*(c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cusq));
      f_PPM = c1o216*(drho+(drho+c1o1)*(c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cusq));
      f_MMP = c1o216*(drho+(drho+c1o1)*(c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cusq));
      f_PMP = c1o216*(drho+(drho+c1o1)*(c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cusq));
      f_MPM = c1o216*(drho+(drho+c1o1)*(c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cusq));
      f_PMM = c1o216*(drho+(drho+c1o1)*(c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cusq));
      f_MPP = c1o216*(drho+(drho+c1o1)*(c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cusq));

      ////////////////////////////////////////////////////////////////////////////////
      //! write the new distributions to the bc nodes
      //!
      (dist.f[DIR_P00])[kP00] = f_M00;
      (dist.f[DIR_PP0])[kPP0] = f_MM0;
      (dist.f[DIR_P0M])[kP0M] = f_M0P;
      (dist.f[DIR_PM0])[kPM0] = f_MP0;
      (dist.f[DIR_PMP])[kPMP] = f_MPM;
      (dist.f[DIR_P0P])[kP0P] = f_M0M;
      (dist.f[DIR_PPM])[kPPM] = f_MMP;
      (dist.f[DIR_PPP])[kPPP] = f_MMM;
      (dist.f[DIR_PMM])[kPMM] = f_MPP;

      (dist.f[DIR_M00])[kM00] = f_P00;
      (dist.f[DIR_MM0])[kMM0] = f_PP0;
      (dist.f[DIR_M0M])[kM0M] = f_P0P;
      (dist.f[DIR_MP0])[kMP0] = f_PM0;
      (dist.f[DIR_M0P])[kM0P] = f_P0M;
      (dist.f[DIR_MMM])[kMMM] = f_PPP;
      (dist.f[DIR_MMP])[kMMP] = f_PPM;
      (dist.f[DIR_MPP])[kMPP] = f_PMM;
      (dist.f[DIR_MPM])[kMPM] = f_PMP;

      (dist.f[DIR_0P0])[k0P0] = f_0M0;
      (dist.f[DIR_0M0])[k0M0] = f_0P0;
      (dist.f[DIR_00P])[k00P] = f_00M;
      (dist.f[DIR_00M])[k00M] = f_00P;
      (dist.f[DIR_0PP])[k0PP] = f_0MM;
      (dist.f[DIR_0MM])[k0MM] = f_0PP;
      (dist.f[DIR_0PM])[k0PM] = f_0MP;
      (dist.f[DIR_0MP])[k0MP] = f_0PM;
      (dist.f[DIR_000])[k000] = f_000;
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////





































////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void PrecursorDeviceDistributions(
    int *subgridDistanceIndices,
    int numberOfBCnodes,
    int numberOfPrecursorNodes,
    real* distributions,
    uint* neighborX,
    uint* neighborY,
    uint* neighborZ,
    uint* neighbors0PP,
    uint* neighbors0PM,
    uint* neighbors0MP,
    uint* neighbors0MM,
    real* weights0PP,
    real* weights0PM,
    real* weights0MP,
    real* weights0MM,
    real* fsLast,
    real* fsNext,
    real timeRatio,
    unsigned long long numberOfLBnodes,
    bool isEvenTimestep)
{
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned nodeIndex = getNodeIndex();

    if(nodeIndex>=numberOfBCnodes) return;

    uint kNeighbor0PP = neighbors0PP[nodeIndex];
    real d0PP = weights0PP[nodeIndex];

    real f0LastInterp, f1LastInterp, f2LastInterp, f3LastInterp, f4LastInterp, f5LastInterp, f6LastInterp, f7LastInterp, f8LastInterp;
    real f0NextInterp, f1NextInterp, f2NextInterp, f3NextInterp, f4NextInterp, f5NextInterp, f6NextInterp, f7NextInterp, f8NextInterp;

    real* f0Last = fsLast;
    real* f1Last = &fsLast[  numberOfPrecursorNodes];
    real* f2Last = &fsLast[2*numberOfPrecursorNodes];
    real* f3Last = &fsLast[3*numberOfPrecursorNodes];
    real* f4Last = &fsLast[4*numberOfPrecursorNodes];
    real* f5Last = &fsLast[5*numberOfPrecursorNodes];
    real* f6Last = &fsLast[6*numberOfPrecursorNodes];
    real* f7Last = &fsLast[7*numberOfPrecursorNodes];
    real* f8Last = &fsLast[8*numberOfPrecursorNodes];

    real* f0Next = fsNext;
    real* f1Next = &fsNext[  numberOfPrecursorNodes];
    real* f2Next = &fsNext[2*numberOfPrecursorNodes];
    real* f3Next = &fsNext[3*numberOfPrecursorNodes];
    real* f4Next = &fsNext[4*numberOfPrecursorNodes];
    real* f5Next = &fsNext[5*numberOfPrecursorNodes];
    real* f6Next = &fsNext[6*numberOfPrecursorNodes];
    real* f7Next = &fsNext[7*numberOfPrecursorNodes];
    real* f8Next = &fsNext[8*numberOfPrecursorNodes];


    if(d0PP<1e6)
    {
        uint kNeighbor0PM = neighbors0PM[nodeIndex];
        uint kNeighbor0MP = neighbors0MP[nodeIndex];
        uint kNeighbor0MM = neighbors0MM[nodeIndex];

        real d0PM = weights0PM[nodeIndex];
        real d0MP = weights0MP[nodeIndex];
        real d0MM = weights0MM[nodeIndex];

        real invWeightSum = 1.f/(d0PP+d0PM+d0MP+d0MM);

        f0LastInterp = (f0Last[kNeighbor0PP]*d0PP + f0Last[kNeighbor0PM]*d0PM + f0Last[kNeighbor0MP]*d0MP + f0Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f0NextInterp = (f0Next[kNeighbor0PP]*d0PP + f0Next[kNeighbor0PM]*d0PM + f0Next[kNeighbor0MP]*d0MP + f0Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f1LastInterp = (f1Last[kNeighbor0PP]*d0PP + f1Last[kNeighbor0PM]*d0PM + f1Last[kNeighbor0MP]*d0MP + f1Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f1NextInterp = (f1Next[kNeighbor0PP]*d0PP + f1Next[kNeighbor0PM]*d0PM + f1Next[kNeighbor0MP]*d0MP + f1Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f2LastInterp = (f2Last[kNeighbor0PP]*d0PP + f2Last[kNeighbor0PM]*d0PM + f2Last[kNeighbor0MP]*d0MP + f2Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f2NextInterp = (f2Next[kNeighbor0PP]*d0PP + f2Next[kNeighbor0PM]*d0PM + f2Next[kNeighbor0MP]*d0MP + f2Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f3LastInterp = (f3Last[kNeighbor0PP]*d0PP + f3Last[kNeighbor0PM]*d0PM + f3Last[kNeighbor0MP]*d0MP + f3Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f3NextInterp = (f3Next[kNeighbor0PP]*d0PP + f3Next[kNeighbor0PM]*d0PM + f3Next[kNeighbor0MP]*d0MP + f3Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f4LastInterp = (f4Last[kNeighbor0PP]*d0PP + f4Last[kNeighbor0PM]*d0PM + f4Last[kNeighbor0MP]*d0MP + f4Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f4NextInterp = (f4Next[kNeighbor0PP]*d0PP + f4Next[kNeighbor0PM]*d0PM + f4Next[kNeighbor0MP]*d0MP + f4Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f5LastInterp = (f5Last[kNeighbor0PP]*d0PP + f5Last[kNeighbor0PM]*d0PM + f5Last[kNeighbor0MP]*d0MP + f5Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f5NextInterp = (f5Next[kNeighbor0PP]*d0PP + f5Next[kNeighbor0PM]*d0PM + f5Next[kNeighbor0MP]*d0MP + f5Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f6LastInterp = (f6Last[kNeighbor0PP]*d0PP + f6Last[kNeighbor0PM]*d0PM + f6Last[kNeighbor0MP]*d0MP + f6Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f6NextInterp = (f6Next[kNeighbor0PP]*d0PP + f6Next[kNeighbor0PM]*d0PM + f6Next[kNeighbor0MP]*d0MP + f6Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f7LastInterp = (f7Last[kNeighbor0PP]*d0PP + f7Last[kNeighbor0PM]*d0PM + f7Last[kNeighbor0MP]*d0MP + f7Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f7NextInterp = (f7Next[kNeighbor0PP]*d0PP + f7Next[kNeighbor0PM]*d0PM + f7Next[kNeighbor0MP]*d0MP + f7Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f8LastInterp = (f8Last[kNeighbor0PP]*d0PP + f8Last[kNeighbor0PM]*d0PM + f8Last[kNeighbor0MP]*d0MP + f8Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f8NextInterp = (f8Next[kNeighbor0PP]*d0PP + f8Next[kNeighbor0PM]*d0PM + f8Next[kNeighbor0MP]*d0MP + f8Next[kNeighbor0MM]*d0MM)*invWeightSum;

    } else {
        f0LastInterp = f0Last[kNeighbor0PP];
        f1LastInterp = f1Last[kNeighbor0PP];
        f2LastInterp = f2Last[kNeighbor0PP];
        f3LastInterp = f3Last[kNeighbor0PP];
        f4LastInterp = f4Last[kNeighbor0PP];
        f5LastInterp = f5Last[kNeighbor0PP];
        f6LastInterp = f6Last[kNeighbor0PP];
        f7LastInterp = f7Last[kNeighbor0PP];
        f8LastInterp = f8Last[kNeighbor0PP];

        f0NextInterp = f0Next[kNeighbor0PP];
        f1NextInterp = f1Next[kNeighbor0PP];
        f2NextInterp = f2Next[kNeighbor0PP];
        f3NextInterp = f3Next[kNeighbor0PP];
        f4NextInterp = f4Next[kNeighbor0PP];
        f5NextInterp = f5Next[kNeighbor0PP];
        f6NextInterp = f6Next[kNeighbor0PP];
        f7NextInterp = f7Next[kNeighbor0PP];
        f8NextInterp = f8Next[kNeighbor0PP];
    }
    //////////////////////////////////////////////////////////////////////////
    //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep
    //! is based on the esoteric twist algorithm \ref <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier
    //! et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
    //!
    Distributions27 dist;
    getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

    unsigned int KQK  = subgridDistanceIndices[nodeIndex];
    // unsigned int k000= KQK;
    unsigned int kP00   = KQK;
    // unsigned int kM00   = neighborX[KQK];
    // unsigned int k0P0   = KQK;
    unsigned int k0M0   = neighborY[KQK];
    // unsigned int k00P   = KQK;
    unsigned int k00M   = neighborZ[KQK];
    // unsigned int kMM0  = neighborY[kM00];
    unsigned int kPP0  = KQK;
    unsigned int kPM0  = k0M0;
    // unsigned int kMP0  = kM00;
    // unsigned int kM0M  = neighborZ[kM00];
    unsigned int kP0P  = KQK;
    unsigned int kP0M  = k00M;
    // unsigned int kM0P  = kM00;
    unsigned int k0MM  = neighborZ[k0M0];
    // unsigned int k0PM  = k00M;
    // unsigned int k0MP  = k0M0;
    unsigned int kPMP = k0M0;
    // unsigned int kMPM = kM0M;
    // unsigned int kMPP = kM00;
    unsigned int kPMM = k0MM;
    // unsigned int kMMP = kMM0;
    unsigned int kPPM = k00M;
    unsigned int kPPP = KQK;
    // unsigned int kMMM = neighborZ[kMM0];

    dist.f[DIR_P00][kP00] = f0LastInterp*(1.f-timeRatio) + f0NextInterp*timeRatio;
    dist.f[DIR_PP0][kPP0] = f1LastInterp*(1.f-timeRatio) + f1NextInterp*timeRatio;
    dist.f[DIR_PM0][kPM0] = f2LastInterp*(1.f-timeRatio) + f2NextInterp*timeRatio;
    dist.f[DIR_P0P][kP0P] = f3LastInterp*(1.f-timeRatio) + f3NextInterp*timeRatio;
    dist.f[DIR_P0M][kP0M] = f4LastInterp*(1.f-timeRatio) + f4NextInterp*timeRatio;
    dist.f[DIR_PPP][kPPP] = f5LastInterp*(1.f-timeRatio) + f5NextInterp*timeRatio;
    dist.f[DIR_PMP][kPMP] = f6LastInterp*(1.f-timeRatio) + f6NextInterp*timeRatio;
    dist.f[DIR_PPM][kPPM] = f7LastInterp*(1.f-timeRatio) + f7NextInterp*timeRatio;
    dist.f[DIR_PMM][kPMM] = f8LastInterp*(1.f-timeRatio) + f8NextInterp*timeRatio;
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////












































////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// NOTE: Has not been tested after bug fix!
__global__ void QPrecursorDeviceDistributions(
    int* subgridDistanceIndices,
    real* subgridDistances,
    int sizeQ,
    int numberOfBCnodes,
    int numberOfPrecursorNodes,
    real* distributions,
    uint* neighborX,
    uint* neighborY,
    uint* neighborZ,
    uint* neighbors0PP,
    uint* neighbors0PM,
    uint* neighbors0MP,
    uint* neighbors0MM,
    real* weights0PP,
    real* weights0PM,
    real* weights0MP,
    real* weights0MM,
    real* fsLast,
    real* fsNext,
    real timeRatio,
    unsigned long long numberOfLBnodes,
    bool isEvenTimestep)
{
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned nodeIndex = getNodeIndex();

    if(nodeIndex>=numberOfBCnodes) return;

    uint kNeighbor0PP = neighbors0PP[nodeIndex];
    real d0PP = weights0PP[nodeIndex];

    real f0LastInterp, f1LastInterp, f2LastInterp, f3LastInterp, f4LastInterp, f5LastInterp, f6LastInterp, f7LastInterp, f8LastInterp;
    real f0NextInterp, f1NextInterp, f2NextInterp, f3NextInterp, f4NextInterp, f5NextInterp, f6NextInterp, f7NextInterp, f8NextInterp;

    real* f0Last = fsLast;
    real* f1Last = &fsLast[  numberOfPrecursorNodes];
    real* f2Last = &fsLast[2*numberOfPrecursorNodes];
    real* f3Last = &fsLast[3*numberOfPrecursorNodes];
    real* f4Last = &fsLast[4*numberOfPrecursorNodes];
    real* f5Last = &fsLast[5*numberOfPrecursorNodes];
    real* f6Last = &fsLast[6*numberOfPrecursorNodes];
    real* f7Last = &fsLast[7*numberOfPrecursorNodes];
    real* f8Last = &fsLast[8*numberOfPrecursorNodes];

    real* f0Next = fsNext;
    real* f1Next = &fsNext[  numberOfPrecursorNodes];
    real* f2Next = &fsNext[2*numberOfPrecursorNodes];
    real* f3Next = &fsNext[3*numberOfPrecursorNodes];
    real* f4Next = &fsNext[4*numberOfPrecursorNodes];
    real* f5Next = &fsNext[5*numberOfPrecursorNodes];
    real* f6Next = &fsNext[6*numberOfPrecursorNodes];
    real* f7Next = &fsNext[7*numberOfPrecursorNodes];
    real* f8Next = &fsNext[8*numberOfPrecursorNodes];


    if(d0PP<1e6)
    {
        uint kNeighbor0PM = neighbors0PM[nodeIndex];
        uint kNeighbor0MP = neighbors0MP[nodeIndex];
        uint kNeighbor0MM = neighbors0MM[nodeIndex];

        real d0PM = weights0PM[nodeIndex];
        real d0MP = weights0MP[nodeIndex];
        real d0MM = weights0MM[nodeIndex];

        real invWeightSum = 1.f/(d0PP+d0PM+d0MP+d0MM);

        f0LastInterp = (f0Last[kNeighbor0PP]*d0PP + f0Last[kNeighbor0PM]*d0PM + f0Last[kNeighbor0MP]*d0MP + f0Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f0NextInterp = (f0Next[kNeighbor0PP]*d0PP + f0Next[kNeighbor0PM]*d0PM + f0Next[kNeighbor0MP]*d0MP + f0Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f1LastInterp = (f1Last[kNeighbor0PP]*d0PP + f1Last[kNeighbor0PM]*d0PM + f1Last[kNeighbor0MP]*d0MP + f1Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f1NextInterp = (f1Next[kNeighbor0PP]*d0PP + f1Next[kNeighbor0PM]*d0PM + f1Next[kNeighbor0MP]*d0MP + f1Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f2LastInterp = (f2Last[kNeighbor0PP]*d0PP + f2Last[kNeighbor0PM]*d0PM + f2Last[kNeighbor0MP]*d0MP + f2Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f2NextInterp = (f2Next[kNeighbor0PP]*d0PP + f2Next[kNeighbor0PM]*d0PM + f2Next[kNeighbor0MP]*d0MP + f2Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f3LastInterp = (f3Last[kNeighbor0PP]*d0PP + f3Last[kNeighbor0PM]*d0PM + f3Last[kNeighbor0MP]*d0MP + f3Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f3NextInterp = (f3Next[kNeighbor0PP]*d0PP + f3Next[kNeighbor0PM]*d0PM + f3Next[kNeighbor0MP]*d0MP + f3Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f4LastInterp = (f4Last[kNeighbor0PP]*d0PP + f4Last[kNeighbor0PM]*d0PM + f4Last[kNeighbor0MP]*d0MP + f4Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f4NextInterp = (f4Next[kNeighbor0PP]*d0PP + f4Next[kNeighbor0PM]*d0PM + f4Next[kNeighbor0MP]*d0MP + f4Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f5LastInterp = (f5Last[kNeighbor0PP]*d0PP + f5Last[kNeighbor0PM]*d0PM + f5Last[kNeighbor0MP]*d0MP + f5Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f5NextInterp = (f5Next[kNeighbor0PP]*d0PP + f5Next[kNeighbor0PM]*d0PM + f5Next[kNeighbor0MP]*d0MP + f5Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f6LastInterp = (f6Last[kNeighbor0PP]*d0PP + f6Last[kNeighbor0PM]*d0PM + f6Last[kNeighbor0MP]*d0MP + f6Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f6NextInterp = (f6Next[kNeighbor0PP]*d0PP + f6Next[kNeighbor0PM]*d0PM + f6Next[kNeighbor0MP]*d0MP + f6Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f7LastInterp = (f7Last[kNeighbor0PP]*d0PP + f7Last[kNeighbor0PM]*d0PM + f7Last[kNeighbor0MP]*d0MP + f7Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f7NextInterp = (f7Next[kNeighbor0PP]*d0PP + f7Next[kNeighbor0PM]*d0PM + f7Next[kNeighbor0MP]*d0MP + f7Next[kNeighbor0MM]*d0MM)*invWeightSum;

        f8LastInterp = (f8Last[kNeighbor0PP]*d0PP + f8Last[kNeighbor0PM]*d0PM + f8Last[kNeighbor0MP]*d0MP + f8Last[kNeighbor0MM]*d0MM)*invWeightSum;
        f8NextInterp = (f8Next[kNeighbor0PP]*d0PP + f8Next[kNeighbor0PM]*d0PM + f8Next[kNeighbor0MP]*d0MP + f8Next[kNeighbor0MM]*d0MM)*invWeightSum;

    } else {
        f0LastInterp = f0Last[kNeighbor0PP];
        f1LastInterp = f1Last[kNeighbor0PP];
        f2LastInterp = f2Last[kNeighbor0PP];
        f3LastInterp = f3Last[kNeighbor0PP];
        f4LastInterp = f4Last[kNeighbor0PP];
        f5LastInterp = f5Last[kNeighbor0PP];
        f6LastInterp = f6Last[kNeighbor0PP];
        f7LastInterp = f7Last[kNeighbor0PP];
        f8LastInterp = f8Last[kNeighbor0PP];

        f0NextInterp = f0Next[kNeighbor0PP];
        f1NextInterp = f1Next[kNeighbor0PP];
        f2NextInterp = f2Next[kNeighbor0PP];
        f3NextInterp = f3Next[kNeighbor0PP];
        f4NextInterp = f4Next[kNeighbor0PP];
        f5NextInterp = f5Next[kNeighbor0PP];
        f6NextInterp = f6Next[kNeighbor0PP];
        f7NextInterp = f7Next[kNeighbor0PP];
        f8NextInterp = f8Next[kNeighbor0PP];
    }
    //////////////////////////////////////////////////////////////////////////
    //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep
    //! is based on the esoteric twist algorithm \ref <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier
    //! et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
    //!
    Distributions27 dist;
    getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

    unsigned int KQK  = subgridDistanceIndices[nodeIndex];
    // unsigned int k000= KQK;
    unsigned int kP00   = KQK;
    // unsigned int kM00   = neighborX[KQK];
    // unsigned int k0P0   = KQK;
    unsigned int k0M0   = neighborY[KQK];
    // unsigned int k00P   = KQK;
    unsigned int k00M   = neighborZ[KQK];
    // unsigned int kMM0  = neighborY[kM00];
    unsigned int kPP0  = KQK;
    unsigned int kPM0  = k0M0;
    // unsigned int kMP0  = kM00;
    // unsigned int kM0M  = neighborZ[kM00];
    unsigned int kP0P  = KQK;
    unsigned int kP0M  = k00M;
    // unsigned int kM0P  = kM00;
    unsigned int k0MM  = neighborZ[k0M0];
    // unsigned int k0PM  = k00M;
    // unsigned int k0MP  = k0M0;
    unsigned int kPMP = k0M0;
    // unsigned int kMPM = kM0M;
    // unsigned int kMPP = kM00;
    unsigned int kPMM = k0MM;
    // unsigned int kMMP = kMM0;
    unsigned int kPPM = k00M;
    unsigned int kPPP = KQK;
    // unsigned int kMMM = neighborZ[kMM0];
    SubgridDistances27 qs;
    getPointersToSubgridDistances(qs, subgridDistances, sizeQ);

    real q;
    q = qs.q[DIR_P00][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_P00][kP00] = f0LastInterp*(1.f-timeRatio) + f0NextInterp*timeRatio;
    q = qs.q[DIR_PP0][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_PP0][kPP0] = f1LastInterp*(1.f-timeRatio) + f1NextInterp*timeRatio;
    q = qs.q[DIR_PM0][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_PM0][kPM0] = f2LastInterp*(1.f-timeRatio) + f2NextInterp*timeRatio;
    q = qs.q[DIR_P0P][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_P0P][kP0P] = f3LastInterp*(1.f-timeRatio) + f3NextInterp*timeRatio;
    q = qs.q[DIR_P0M][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_P0M][kP0M] = f4LastInterp*(1.f-timeRatio) + f4NextInterp*timeRatio;
    q = qs.q[DIR_PPP][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_PPP][kPPP] = f5LastInterp*(1.f-timeRatio) + f5NextInterp*timeRatio;
    q = qs.q[DIR_PMP][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_PMP][kPMP] = f6LastInterp*(1.f-timeRatio) + f6NextInterp*timeRatio;
    q = qs.q[DIR_PPM][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_PPM][kPPM] = f7LastInterp*(1.f-timeRatio) + f7NextInterp*timeRatio;
    q = qs.q[DIR_PMM][nodeIndex]; if(q>= c0o1 && q <= c1o1) dist.f[DIR_PMM][kPMM] = f8LastInterp*(1.f-timeRatio) + f8NextInterp*timeRatio;

}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
