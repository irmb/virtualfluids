#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __         
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |        
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |        
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |        
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____    
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|   
//      \    \  |    |   ________________________________________________________________    
//       \    \ |    |  |  ______________________________________________________________|   
//        \    \|    |  |  |         __          __     __     __     ______      _______    
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)   
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______    
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/   
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can 
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of 
//  the License, or (at your option) any later version.
//  
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT 
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or 
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License 
//  for more details.
//  
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file Cumulant27chim.cu
//! \ingroup GPU
//! \author Martin Schoenherr
//=======================================================================================
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include "math.h"

#include <lbm/CumulantChimeraPreCompiled.h>
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;


__device__ Distributions27 getDistributions27(real* distributions, unsigned int size_Mat, bool isEvenTimestep)
{
    Distributions27 dist;
    if (isEvenTimestep)
    {
        dist.f[dirE   ] = &distributions[dirE   *size_Mat];
        dist.f[dirW   ] = &distributions[dirW   *size_Mat];
        dist.f[dirN   ] = &distributions[dirN   *size_Mat];
        dist.f[dirS   ] = &distributions[dirS   *size_Mat];
        dist.f[dirT   ] = &distributions[dirT   *size_Mat];
        dist.f[dirB   ] = &distributions[dirB   *size_Mat];
        dist.f[dirNE  ] = &distributions[dirNE  *size_Mat];
        dist.f[dirSW  ] = &distributions[dirSW  *size_Mat];
        dist.f[dirSE  ] = &distributions[dirSE  *size_Mat];
        dist.f[dirNW  ] = &distributions[dirNW  *size_Mat];
        dist.f[dirTE  ] = &distributions[dirTE  *size_Mat];
        dist.f[dirBW  ] = &distributions[dirBW  *size_Mat];
        dist.f[dirBE  ] = &distributions[dirBE  *size_Mat];
        dist.f[dirTW  ] = &distributions[dirTW  *size_Mat];
        dist.f[dirTN  ] = &distributions[dirTN  *size_Mat];
        dist.f[dirBS  ] = &distributions[dirBS  *size_Mat];
        dist.f[dirBN  ] = &distributions[dirBN  *size_Mat];
        dist.f[dirTS  ] = &distributions[dirTS  *size_Mat];
        dist.f[dirREST] = &distributions[dirREST*size_Mat];
        dist.f[dirTNE ] = &distributions[dirTNE *size_Mat];
        dist.f[dirTSW ] = &distributions[dirTSW *size_Mat];
        dist.f[dirTSE ] = &distributions[dirTSE *size_Mat];
        dist.f[dirTNW ] = &distributions[dirTNW *size_Mat];
        dist.f[dirBNE ] = &distributions[dirBNE *size_Mat];
        dist.f[dirBSW ] = &distributions[dirBSW *size_Mat];
        dist.f[dirBSE ] = &distributions[dirBSE *size_Mat];
        dist.f[dirBNW ] = &distributions[dirBNW *size_Mat];
    }
    else
    {
        dist.f[dirW   ] = &distributions[dirE   *size_Mat];
        dist.f[dirE   ] = &distributions[dirW   *size_Mat];
        dist.f[dirS   ] = &distributions[dirN   *size_Mat];
        dist.f[dirN   ] = &distributions[dirS   *size_Mat];
        dist.f[dirB   ] = &distributions[dirT   *size_Mat];
        dist.f[dirT   ] = &distributions[dirB   *size_Mat];
        dist.f[dirSW  ] = &distributions[dirNE  *size_Mat];
        dist.f[dirNE  ] = &distributions[dirSW  *size_Mat];
        dist.f[dirNW  ] = &distributions[dirSE  *size_Mat];
        dist.f[dirSE  ] = &distributions[dirNW  *size_Mat];
        dist.f[dirBW  ] = &distributions[dirTE  *size_Mat];
        dist.f[dirTE  ] = &distributions[dirBW  *size_Mat];
        dist.f[dirTW  ] = &distributions[dirBE  *size_Mat];
        dist.f[dirBE  ] = &distributions[dirTW  *size_Mat];
        dist.f[dirBS  ] = &distributions[dirTN  *size_Mat];
        dist.f[dirTN  ] = &distributions[dirBS  *size_Mat];
        dist.f[dirTS  ] = &distributions[dirBN  *size_Mat];
        dist.f[dirBN  ] = &distributions[dirTS  *size_Mat];
        dist.f[dirREST] = &distributions[dirREST*size_Mat];
        dist.f[dirBSW ] = &distributions[dirTNE *size_Mat];
        dist.f[dirBNE ] = &distributions[dirTSW *size_Mat];
        dist.f[dirBNW ] = &distributions[dirTSE *size_Mat];
        dist.f[dirBSE ] = &distributions[dirTNW *size_Mat];
        dist.f[dirTSW ] = &distributions[dirBNE *size_Mat];
        dist.f[dirTNE ] = &distributions[dirBSW *size_Mat];
        dist.f[dirTNW ] = &distributions[dirBSE *size_Mat];
        dist.f[dirTSE ] = &distributions[dirBNW *size_Mat];
    }
    return dist;
}

struct DistributionWrapper
{
    __device__ DistributionWrapper(
        real* distributions,
        unsigned int size_Mat,
        bool isEvenTimestep,
        uint k,
        uint* neighborX,
        uint* neighborY,
        uint* neighborZ) :
        dist(getDistributions27(distributions, size_Mat, isEvenTimestep)),
        k(k),
        kw  (neighborX[k]),
        ks  (neighborY[k]),
        kb  (neighborZ[k]),
        ksw (neighborY[kw]),
        kbw (neighborZ[kw]),
        kbs (neighborZ[ks]),
        kbsw(neighborZ[ksw])
    { 
        read();
    }

    __device__ void read()
    {
        distribution.f[vf::lbm::dir::PZZ] = (dist.f[dirE   ])[k];
        distribution.f[vf::lbm::dir::MZZ] = (dist.f[dirW   ])[kw];
        distribution.f[vf::lbm::dir::ZPZ] = (dist.f[dirN   ])[k];
        distribution.f[vf::lbm::dir::ZMZ] = (dist.f[dirS   ])[ks];
        distribution.f[vf::lbm::dir::ZZP] = (dist.f[dirT   ])[k];
        distribution.f[vf::lbm::dir::ZZM] = (dist.f[dirB   ])[kb];
        distribution.f[vf::lbm::dir::PPZ] = (dist.f[dirNE  ])[k];
        distribution.f[vf::lbm::dir::MMZ] = (dist.f[dirSW  ])[ksw];
        distribution.f[vf::lbm::dir::PMZ] = (dist.f[dirSE  ])[ks];
        distribution.f[vf::lbm::dir::MPZ] = (dist.f[dirNW  ])[kw];
        distribution.f[vf::lbm::dir::PZP] = (dist.f[dirTE  ])[k];
        distribution.f[vf::lbm::dir::MZM] = (dist.f[dirBW  ])[kbw];
        distribution.f[vf::lbm::dir::PZM] = (dist.f[dirBE  ])[kb];
        distribution.f[vf::lbm::dir::MZP] = (dist.f[dirTW  ])[kw];
        distribution.f[vf::lbm::dir::ZPP] = (dist.f[dirTN  ])[k];
        distribution.f[vf::lbm::dir::ZMM] = (dist.f[dirBS  ])[kbs];
        distribution.f[vf::lbm::dir::ZPM] = (dist.f[dirBN  ])[kb];
        distribution.f[vf::lbm::dir::ZMP] = (dist.f[dirTS  ])[ks];
        distribution.f[vf::lbm::dir::PPP] = (dist.f[dirTNE ])[k];
        distribution.f[vf::lbm::dir::MPP] = (dist.f[dirTNW ])[kw];
        distribution.f[vf::lbm::dir::PMP] = (dist.f[dirTSE ])[ks];
        distribution.f[vf::lbm::dir::MMP] = (dist.f[dirTSW ])[ksw];
        distribution.f[vf::lbm::dir::PPM] = (dist.f[dirBNE ])[kb];
        distribution.f[vf::lbm::dir::MPM] = (dist.f[dirBNW ])[kbw];
        distribution.f[vf::lbm::dir::PMM] = (dist.f[dirBSE ])[kbs];
        distribution.f[vf::lbm::dir::MMM] = (dist.f[dirBSW ])[kbsw];
        distribution.f[vf::lbm::dir::ZZZ] = (dist.f[dirREST])[k];
    }

    __device__ void write()
    {
        (dist.f[dirE   ])[k]    = distribution.f[vf::lbm::dir::PZZ];
        (dist.f[dirW   ])[kw]   = distribution.f[vf::lbm::dir::MZZ];
        (dist.f[dirN   ])[k]    = distribution.f[vf::lbm::dir::ZPZ];
        (dist.f[dirS   ])[ks]   = distribution.f[vf::lbm::dir::ZMZ];
        (dist.f[dirT   ])[k]    = distribution.f[vf::lbm::dir::ZZP];
        (dist.f[dirB   ])[kb]   = distribution.f[vf::lbm::dir::ZZM];
        (dist.f[dirNE  ])[k]    = distribution.f[vf::lbm::dir::PPZ];
        (dist.f[dirSW  ])[ksw]  = distribution.f[vf::lbm::dir::MMZ];
        (dist.f[dirSE  ])[ks]   = distribution.f[vf::lbm::dir::PMZ];
        (dist.f[dirNW  ])[kw]   = distribution.f[vf::lbm::dir::MPZ];
        (dist.f[dirTE  ])[k]    = distribution.f[vf::lbm::dir::PZP];
        (dist.f[dirBW  ])[kbw]  = distribution.f[vf::lbm::dir::MZM];
        (dist.f[dirBE  ])[kb]   = distribution.f[vf::lbm::dir::PZM];
        (dist.f[dirTW  ])[kw]   = distribution.f[vf::lbm::dir::MZP];
        (dist.f[dirTN  ])[k]    = distribution.f[vf::lbm::dir::ZPP];
        (dist.f[dirBS  ])[kbs]  = distribution.f[vf::lbm::dir::ZMM];
        (dist.f[dirBN  ])[kb]   = distribution.f[vf::lbm::dir::ZPM];
        (dist.f[dirTS  ])[ks]   = distribution.f[vf::lbm::dir::ZMP];
        (dist.f[dirTNE ])[k]    = distribution.f[vf::lbm::dir::PPP];
        (dist.f[dirTNW ])[kw]   = distribution.f[vf::lbm::dir::MPP];
        (dist.f[dirTSE ])[ks]   = distribution.f[vf::lbm::dir::PMP];
        (dist.f[dirTSW ])[ksw]  = distribution.f[vf::lbm::dir::MMP];
        (dist.f[dirBNE ])[kb]   = distribution.f[vf::lbm::dir::PPM];
        (dist.f[dirBNW ])[kbw]  = distribution.f[vf::lbm::dir::MPM];
        (dist.f[dirBSE ])[kbs]  = distribution.f[vf::lbm::dir::PMM];
        (dist.f[dirBSW ])[kbsw] = distribution.f[vf::lbm::dir::MMM];
        (dist.f[dirREST])[k]    = distribution.f[vf::lbm::dir::ZZZ];
    }

    Distributions27 dist;

    vf::lbm::Distribution27 distribution;

    const uint k;
    const uint kw;
    const uint ks;
    const uint kb;
    const uint ksw;
    const uint kbw;
    const uint kbs;
    const uint kbsw;
};

__device__ unsigned int getNodeIndex()
{
    const unsigned  x = threadIdx.x; 
    const unsigned  y = blockIdx.x;  
    const unsigned  z = blockIdx.y;  

    const unsigned nx = blockDim.x;
    const unsigned ny = gridDim.x;

    return nx*(ny*z + y) + x;
}

__device__ bool isValidFluidNode(uint k, int size_Mat, uint nodeType)
{
    return (k < size_Mat) && (nodeType == GEO_FLUID);
}

__device__ void getLevelForce(real fx, real fy, real fz, int level, real* forces)
{
    real fx_t {1.}, fy_t {1.}, fz_t {1.};
    for (int i = 0; i < level; i++)
    {
        fx_t *= c2o1;
        fy_t *= c2o1;
        fz_t *= c2o1;
    }

    forces[0] = fx / fx_t;
    forces[1] = fy / fy_t;
    forces[2] = fz / fz_t;
}


extern "C" __global__ void Cumulant_K17_LBM_Device_Kernel(
    real omega,
    uint* typeOfGridNode,
    uint* neighborX,
    uint* neighborY,
    uint* neighborZ,
    real* distributions,
    int size_Mat,
    int level,
    real* forces,
    bool isEvenTimestep)
{
    const uint k = getNodeIndex();
    const uint nodeType = typeOfGridNode[k];

    if (isValidFluidNode(k, size_Mat, nodeType))
    {
        DistributionWrapper distributionWrapper {
            distributions, size_Mat, isEvenTimestep, k, neighborX, neighborY, neighborZ
        };

        real level_forces[3];
        getLevelForce(forces[0], forces[1], forces[2], level, level_forces);

        vf::lbm::cumulantChimera(distributionWrapper.distribution, omega, level_forces);

        distributionWrapper.write();
    }
}



////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void Cumulant_K17_LBM_Device_Kernel_old(
    real omega,
    uint* typeOfGridNode,
    uint* neighborX,
    uint* neighborY,
    uint* neighborZ,
    real* distributions,
    int size_Mat,
    int level,
    real* forces,
    bool isEvenTimestep)
{
    //////////////////////////////////////////////////////////////////////////
    //! Cumulant K17 Kernel is based on \ref
    //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05.040 ]</b></a>
    //! and \ref
    //! <a href="https://doi.org/10.1016/j.jcp.2017.07.004"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.07.004 ]</b></a>
    //!
    //! The cumulant kernel is executed in the following steps
    //!
    ////////////////////////////////////////////////////////////////////////////////
    //! - Get node index coordinates from thredIdx, blockIdx, blockDim and gridDim.
    //!
    const unsigned  x = threadIdx.x; 
    const unsigned  y = blockIdx.x;  
    const unsigned  z = blockIdx.y;  

    const unsigned nx = blockDim.x;
    const unsigned ny = gridDim.x;

    const unsigned k = nx*(ny*z + y) + x;

    //////////////////////////////////////////////////////////////////////////
    // run for all indices in size_Mat and fluid nodes
    if ((k < size_Mat) && (typeOfGridNode[k] == GEO_FLUID))
    {
        //////////////////////////////////////////////////////////////////////////
        //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
        //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
        //!
        Distributions27 dist = getDistributions27(distributions, size_Mat, isEvenTimestep);

        ////////////////////////////////////////////////////////////////////////////////
        //! - Set neighbor indices (necessary for indirect addressing) 
        uint kw   = neighborX[k];
        uint ks   = neighborY[k];
        uint kb   = neighborZ[k];
        uint ksw  = neighborY[kw];
        uint kbw  = neighborZ[kw];
        uint kbs  = neighborZ[ks];
        uint kbsw = neighborZ[ksw];
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Set local distributions
        //!
        real mfcbb = (dist.f[dirE   ])[k];
        real mfabb = (dist.f[dirW   ])[kw];
        real mfbcb = (dist.f[dirN   ])[k];
        real mfbab = (dist.f[dirS   ])[ks];
        real mfbbc = (dist.f[dirT   ])[k];
        real mfbba = (dist.f[dirB   ])[kb];
        real mfccb = (dist.f[dirNE  ])[k];
        real mfaab = (dist.f[dirSW  ])[ksw];
        real mfcab = (dist.f[dirSE  ])[ks];
        real mfacb = (dist.f[dirNW  ])[kw];
        real mfcbc = (dist.f[dirTE  ])[k];
        real mfaba = (dist.f[dirBW  ])[kbw];
        real mfcba = (dist.f[dirBE  ])[kb];
        real mfabc = (dist.f[dirTW  ])[kw];
        real mfbcc = (dist.f[dirTN  ])[k];
        real mfbaa = (dist.f[dirBS  ])[kbs];
        real mfbca = (dist.f[dirBN  ])[kb];
        real mfbac = (dist.f[dirTS  ])[ks];
        real mfbbb = (dist.f[dirREST])[k];
        real mfccc = (dist.f[dirTNE ])[k];
        real mfaac = (dist.f[dirTSW ])[ksw];
        real mfcac = (dist.f[dirTSE ])[ks];
        real mfacc = (dist.f[dirTNW ])[kw];
        real mfcca = (dist.f[dirBNE ])[kb];
        real mfaaa = (dist.f[dirBSW ])[kbsw];
        real mfcaa = (dist.f[dirBSE ])[kbs];
        real mfaca = (dist.f[dirBNW ])[kbw];

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Calculate density and velocity using pyramid summation for low round-off errors as in Eq. (J1)-(J3) \ref
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
        //!
        real drho =
            ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
            (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
            ((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb; 
        real rho = c1o1 + drho;
        real OOrho = c1o1 / rho;    
        real vvx = 
            ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
            (((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
            (mfcbb - mfabb)) * OOrho;
        real vvy = 
            ((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
            (((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
            (mfbcb - mfbab)) * OOrho;
        real vvz = 
            ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
            (((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
            (mfbbc - mfbba)) * OOrho;
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Add half of the acceleration (body force) to the velocity as in Eq. (42) \ref
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
        //!
        real fx = forces[0];
        real fy = forces[1];
        real fz = forces[2];

        real fx_t {1.}, fy_t {1.}, fz_t {1.};
        for (int i = 0; i < level; i++)
        {
            fx_t *= c2o1;
            fy_t *= c2o1;
            fz_t *= c2o1;
        }

        fx /= fx_t;
        fy /= fy_t;
        fz /= fz_t;
        //real forces[3] {fx, fy, fz};

        vvx += fx * c1o2;
        vvy += fy * c1o2;
        vvz += fz * c1o2;
        ////////////////////////////////////////////////////////////////////////////////////
        // calculate the square of velocities for this lattice node
        real vx2 = vvx*vvx;
        real vy2 = vvy*vvy;
        real vz2 = vvz*vvz;
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Set relaxation limiters for third order cumulants to default value \f$ \lambda=0.001 \f$ according to section 6 in \ref
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //!
        real wadjust;
        real qudricLimitP = c1o100;
        real qudricLimitM = c1o100;
        real qudricLimitD = c1o100;
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Chimera transform from well conditioned distributions to central moments as defined in Appendix J in \ref
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
        //! see also Eq. (6)-(14) in \ref
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //!
        ////////////////////////////////////////////////////////////////////////////////////
        // Z - Dir
        vf::lbm::forwardInverseChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, c36o1, c1o36);
        vf::lbm::forwardInverseChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2,  c9o1,  c1o9);
        vf::lbm::forwardInverseChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c36o1, c1o36);
        vf::lbm::forwardInverseChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2,  c9o1,  c1o9);
        vf::lbm::forwardInverseChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2,  c9o4,  c4o9);
        vf::lbm::forwardInverseChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2,  c9o1,  c1o9);
        vf::lbm::forwardInverseChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c36o1, c1o36);
        vf::lbm::forwardInverseChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2,  c9o1,  c1o9);
        vf::lbm::forwardInverseChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c36o1, c1o36);   
        ////////////////////////////////////////////////////////////////////////////////////
        // Y - Dir
        vf::lbm::forwardInverseChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2,  c6o1,  c1o6);
        vf::lbm::forwardChimera(            mfaab, mfabb, mfacb, vvy, vy2);
        vf::lbm::forwardInverseChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c18o1, c1o18);
        vf::lbm::forwardInverseChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2,  c3o2,  c2o3);
        vf::lbm::forwardChimera(            mfbab, mfbbb, mfbcb, vvy, vy2);
        vf::lbm::forwardInverseChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2,  c9o2,  c2o9);
        vf::lbm::forwardInverseChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2,  c6o1,  c1o6);
        vf::lbm::forwardChimera(            mfcab, mfcbb, mfccb, vvy, vy2);
        vf::lbm::forwardInverseChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c18o1, c1o18);   
        ////////////////////////////////////////////////////////////////////////////////////
        // X - Dir
        vf::lbm::forwardInverseChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o1, c1o1);
        vf::lbm::forwardChimera(            mfaba, mfbba, mfcba, vvx, vx2);
        vf::lbm::forwardInverseChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c3o1, c1o3);
        vf::lbm::forwardChimera(            mfaab, mfbab, mfcab, vvx, vx2);
        vf::lbm::forwardChimera(            mfabb, mfbbb, mfcbb, vvx, vx2);
        vf::lbm::forwardChimera(            mfacb, mfbcb, mfccb, vvx, vx2);
        vf::lbm::forwardInverseChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c3o1, c1o3);
        vf::lbm::forwardChimera(            mfabc, mfbbc, mfcbc, vvx, vx2);
        vf::lbm::forwardInverseChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c3o1, c1o9); 
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Setting relaxation rates for non-hydrodynamic cumulants (default values). Variable names and equations    according to
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //!  => [NAME IN PAPER]=[NAME IN CODE]=[DEFAULT VALUE].
        //!  - Trace of second order cumulants \f$ C_{200}+C_{020}+C_{002} \f$ used to adjust bulk  viscosity:\f$\omega_2=OxxPyyPzz=1.0 \f$.
        //!  - Third order cumulants \f$ C_{120}+C_{102}, C_{210}+C_{012}, C_{201}+C_{021} \f$: \f$ \omega_3=OxyyPxzz   \f$ set according to Eq. (111) with simplifications assuming \f$ \omega_2=1.0\f$.
        //!  - Third order cumulants \f$ C_{120}-C_{102}, C_{210}-C_{012}, C_{201}-C_{021} \f$: \f$ \omega_4 =  OxyyMxzz \f$ set according to Eq. (112) with simplifications assuming \f$ \omega_2 = 1.0\f$.
        //!  - Third order cumulants \f$ C_{111} \f$: \f$ \omega_5 = Oxyz \f$ set according to Eq. (113) with   simplifications assuming \f$ \omega_2 = 1.0\f$  (modify for different bulk viscosity).
        //!  - Fourth order cumulants \f$ C_{220}, C_{202}, C_{022}, C_{211}, C_{121}, C_{112} \f$: for simplification  all set to the same default value \f$ \omega_6=\omega_7=\omega_8=O4=1.0 \f$.
        //!  - Fifth order cumulants \f$ C_{221}, C_{212}, C_{122}\f$: \f$\omega_9=O5=1.0\f$.
        //!  - Sixth order cumulant \f$ C_{222}\f$: \f$\omega_{10}=O6=1.0\f$.
        //!
        ////////////////////////////////////////////////////////////
        //2.
        real OxxPyyPzz = c1o1;
        ////////////////////////////////////////////////////////////
        //3.
        real OxyyPxzz = c8o1  * (-c2o1 + omega) * ( c1o1 + c2o1*omega) / (-c8o1 - c14o1*omega + c7o1*omega*omega);
        real OxyyMxzz = c8o1  * (-c2o1 + omega) * (-c7o1 + c4o1*omega) / (c56o1 - c50o1*omega + c9o1*omega*omega);
        real Oxyz     = c24o1 * (-c2o1 + omega) * (-c2o1 - c7o1*omega + c3o1*omega*omega) / (c48o1 + c152o1*omega - c130o1*omega*omega + c29o1*omega*omega*omega);
        ////////////////////////////////////////////////////////////
        //4.
        real O4 = c1o1;
        ////////////////////////////////////////////////////////////
        //5.
        real O5 = c1o1;
        ////////////////////////////////////////////////////////////
        //6.
        real O6 = c1o1; 
        ////////////////////////////////////////////////////////////////////////////////////
        //! - A and B: parameters for fourth order convergence of the diffusion term according to Eq. (114) and (115) 
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //! with simplifications assuming \f$ \omega_2 = 1.0 \f$ (modify for different bulk viscosity).
        //!
        real A = (c4o1 + c2o1*omega - c3o1*omega*omega) / (c2o1 - c7o1*omega + c5o1*omega*omega);
        real B = (c4o1 + c28o1*omega - c14o1*omega*omega) / (c6o1 - c21o1*omega + c15o1*omega*omega);   
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Compute cumulants from central moments according to Eq. (20)-(23) in
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //!
        ////////////////////////////////////////////////////////////
        //4.
        real CUMcbb = mfcbb - ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) * OOrho;
        real CUMbcb = mfbcb - ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) * OOrho;
        real CUMbbc = mfbbc - ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) * OOrho;  
        real CUMcca = mfcca - (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) * OOrho - c1o9*(drho   * OOrho));
        real CUMcac = mfcac - (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) * OOrho - c1o9*(drho   * OOrho));
        real CUMacc = mfacc - (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) * OOrho - c1o9*(drho   * OOrho));
        ////////////////////////////////////////////////////////////
        //5.
        real CUMbcc = mfbcc - ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba *  mfabc)) + c1o3 * (mfbca + mfbac)) * OOrho;
        real CUMcbc = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba *  mfbac)) + c1o3 * (mfcba + mfabc)) * OOrho;
        real CUMccb = mfccb - ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb *  mfcba)) + c1o3 * (mfacb + mfcab)) * OOrho;
        ////////////////////////////////////////////////////////////
        //6.
        real CUMccc = mfccc + ((-c4o1 *  mfbbb * mfbbb
            - (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
            - c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
            - c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) * OOrho
            + (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
            + c2o1 * (mfcaa * mfaca * mfaac)
            + c16o1 *  mfbba * mfbab * mfabb) * OOrho * OOrho
            - c1o3 * (mfacc + mfcac + mfcca) * OOrho
            - c1o9 * (mfcaa + mfaca + mfaac) * OOrho
            + (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
            + (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) * OOrho * OOrho  * c2o3
            + c1o27*((drho * drho - drho) * OOrho * OOrho));    
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Compute linear combinations of second and third order cumulants
        //!
        ////////////////////////////////////////////////////////////
        //2.
        real mxxPyyPzz = mfcaa + mfaca + mfaac;
        real mxxMyy = mfcaa - mfaca;
        real mxxMzz = mfcaa - mfaac;
        ////////////////////////////////////////////////////////////
        //3.
        real mxxyPyzz = mfcba + mfabc;
        real mxxyMyzz = mfcba - mfabc;  
        real mxxzPyyz = mfcab + mfacb;
        real mxxzMyyz = mfcab - mfacb;  
        real mxyyPxzz = mfbca + mfbac;
        real mxyyMxzz = mfbca - mfbac;  
        ////////////////////////////////////////////////////////////////////////////////////
        //incl. correction
        ////////////////////////////////////////////////////////////
        //! - Compute velocity  gradients from second order cumulants according to Eq. (27)-(32)
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //! Further explanations of the correction in viscosity in Appendix H of
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
        //! Note that the division by rho is omitted here as we need rho times the gradients later.
        //!
        real Dxy = -c3o1*omega*mfbba;
        real Dxz = -c3o1*omega*mfbab;
        real Dyz = -c3o1*omega*mfabb;
        real dxux = c1o2 * (-omega) *(mxxMyy + mxxMzz) + c1o2 *  OxxPyyPzz * (mfaaa - mxxPyyPzz);
        real dyuy = dxux + omega * c3o2 * mxxMyy;
        real dzuz = dxux + omega * c3o2 * mxxMzz;
        ////////////////////////////////////////////////////////////
        //! - Relaxation of second order cumulants with correction terms according to Eq. (33)-(35) in
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //!
        mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz) - c3o1 * (c1o1 - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2  * dzuz);
        mxxMyy    += omega * (-mxxMyy) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy);
        mxxMzz    += omega * (-mxxMzz) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz);   
        ////////////////////////////////////////////////////////////////////////////////////
        ////no correction
        //mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz);
        //mxxMyy += -(-omega) * (-mxxMyy);
        //mxxMzz += -(-omega) * (-mxxMzz);
        //////////////////////////////////////////////////////////////////////////
        mfabb += omega * (-mfabb);
        mfbab += omega * (-mfbab);
        mfbba += omega * (-mfbba);  
        ////////////////////////////////////////////////////////////////////////////////////
        //relax
        //////////////////////////////////////////////////////////////////////////
        // incl. limiter
        //! - Relaxation of third order cumulants including limiter according to Eq. (116)-(123)
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //!
        wadjust   = Oxyz + (c1o1 - Oxyz)*abs(mfbbb) / (abs(mfbbb) + qudricLimitD);
        mfbbb    += wadjust * (-mfbbb);
        wadjust   = OxyyPxzz + (c1o1 - OxyyPxzz)*abs(mxxyPyzz) / (abs(mxxyPyzz) + qudricLimitP);
        mxxyPyzz += wadjust * (-mxxyPyzz);
        wadjust   = OxyyMxzz + (c1o1 - OxyyMxzz)*abs(mxxyMyzz) / (abs(mxxyMyzz) + qudricLimitM);
        mxxyMyzz += wadjust * (-mxxyMyzz);
        wadjust   = OxyyPxzz + (c1o1 - OxyyPxzz)*abs(mxxzPyyz) / (abs(mxxzPyyz) + qudricLimitP);
        mxxzPyyz += wadjust * (-mxxzPyyz);
        wadjust   = OxyyMxzz + (c1o1 - OxyyMxzz)*abs(mxxzMyyz) / (abs(mxxzMyyz) + qudricLimitM);
        mxxzMyyz += wadjust * (-mxxzMyyz);
        wadjust   = OxyyPxzz + (c1o1 - OxyyPxzz)*abs(mxyyPxzz) / (abs(mxyyPxzz) + qudricLimitP);
        mxyyPxzz += wadjust * (-mxyyPxzz);
        wadjust   = OxyyMxzz + (c1o1 - OxyyMxzz)*abs(mxyyMxzz) / (abs(mxyyMxzz) + qudricLimitM);
        mxyyMxzz += wadjust * (-mxyyMxzz);
        //////////////////////////////////////////////////////////////////////////
        // no limiter
        //mfbbb += OxyyMxzz * (-mfbbb);
        //mxxyPyzz += OxyyPxzz * (-mxxyPyzz);
        //mxxyMyzz += OxyyMxzz * (-mxxyMyzz);
        //mxxzPyyz += OxyyPxzz * (-mxxzPyyz);
        //mxxzMyyz += OxyyMxzz * (-mxxzMyyz);
        //mxyyPxzz += OxyyPxzz * (-mxyyPxzz);
        //mxyyMxzz += OxyyMxzz * (-mxyyMxzz);   
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Compute inverse linear combinations of second and third order cumulants
        //!
        mfcaa = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
        mfaca = c1o3 * (-c2o1*  mxxMyy + mxxMzz + mxxPyyPzz);
        mfaac = c1o3 * (mxxMyy - c2o1* mxxMzz + mxxPyyPzz); 
        mfcba = ( mxxyMyzz + mxxyPyzz) * c1o2;
        mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
        mfcab = ( mxxzMyyz + mxxzPyyz) * c1o2;
        mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
        mfbca = ( mxyyMxzz + mxyyPxzz) * c1o2;
        mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;
        //////////////////////////////////////////////////////////////////////////  
        //////////////////////////////////////////////////////////////////////////
        //4.
        // no limiter
        //! - Relax fourth order cumulants to modified equilibrium for fourth order convergence of diffusion according  to Eq. (43)-(48)
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //!
        CUMacc = -O4*(c1o1 / omega - c1o2) * (dyuy + dzuz) * c2o3 * A + (c1o1 - O4) * (CUMacc);
        CUMcac = -O4*(c1o1 / omega - c1o2) * (dxux + dzuz) * c2o3 * A + (c1o1 - O4) * (CUMcac);
        CUMcca = -O4*(c1o1 / omega - c1o2) * (dyuy + dxux) * c2o3 * A + (c1o1 - O4) * (CUMcca);
        CUMbbc = -O4*(c1o1 / omega - c1o2) * Dxy           * c1o3 * B + (c1o1 - O4) * (CUMbbc);
        CUMbcb = -O4*(c1o1 / omega - c1o2) * Dxz           * c1o3 * B + (c1o1 - O4) * (CUMbcb);
        CUMcbb = -O4*(c1o1 / omega - c1o2) * Dyz           * c1o3 * B + (c1o1 - O4) * (CUMcbb); 
        //////////////////////////////////////////////////////////////////////////
        //5.
        CUMbcc += O5 * (-CUMbcc);
        CUMcbc += O5 * (-CUMcbc);
        CUMccb += O5 * (-CUMccb);   
        //////////////////////////////////////////////////////////////////////////
        //6.
        CUMccc += O6 * (-CUMccc);   
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Compute central moments from post collision cumulants according to Eq. (53)-(56) in
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //! 
        //////////////////////////////////////////////////////////////////////////
        //4.
        mfcbb = CUMcbb + c1o3*((c3o1*mfcaa + c1o1) * mfabb + c6o1 * mfbba * mfbab) * OOrho;
        mfbcb = CUMbcb + c1o3*((c3o1*mfaca + c1o1) * mfbab + c6o1 * mfbba * mfabb) * OOrho;
        mfbbc = CUMbbc + c1o3*((c3o1*mfaac + c1o1) * mfbba + c6o1 * mfbab * mfabb) * OOrho; 
        mfcca = CUMcca + (((mfcaa * mfaca + c2o1 * mfbba * mfbba)*c9o1 + c3o1 * (mfcaa + mfaca)) * OOrho - (drho *  OOrho))*c1o9;
        mfcac = CUMcac + (((mfcaa * mfaac + c2o1 * mfbab * mfbab)*c9o1 + c3o1 * (mfcaa + mfaac)) * OOrho - (drho *  OOrho))*c1o9;
        mfacc = CUMacc + (((mfaac * mfaca + c2o1 * mfabb * mfabb)*c9o1 + c3o1 * (mfaac + mfaca)) * OOrho - (drho *  OOrho))*c1o9; 
        //////////////////////////////////////////////////////////////////////////
        //5.
        mfbcc = CUMbcc + c1o3 *(c3o1*(mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb +    mfbba * mfabc)) + (mfbca + mfbac)) * OOrho;
        mfcbc = CUMcbc + c1o3 *(c3o1*(mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab +    mfbba * mfbac)) + (mfcba + mfabc)) * OOrho;
        mfccb = CUMccb + c1o3 *(c3o1*(mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca +    mfabb * mfcba)) + (mfacb + mfcab)) * OOrho; 
        //////////////////////////////////////////////////////////////////////////
        //6.
        mfccc =	CUMccc - ((-c4o1 *  mfbbb * mfbbb
                - (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
                - c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
                - c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) * OOrho
                + (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
                    + c2o1 * (mfcaa * mfaca * mfaac)
                    + c16o1 *  mfbba * mfbab * mfabb) * OOrho * OOrho
                - c1o3 * (mfacc + mfcac + mfcca) * OOrho
                - c1o9 * (mfcaa + mfaca + mfaac) * OOrho
                + (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
                    + (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) * OOrho * OOrho * c2o3
                + c1o27*((drho * drho - drho) * OOrho * OOrho));    
        ////////////////////////////////////////////////////////////////////////////////////
        //! -  Add acceleration (body force) to first order cumulants according to Eq. (85)-(87) in
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
        //!
        mfbaa = -mfbaa;
        mfaba = -mfaba;
        mfaab = -mfaab; 
        ////////////////////////////////////////////////////////////////////////////////////
        //! - Chimera transform from central moments to well conditioned distributions as defined in Appendix J in
        //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa  2015.05.001 ]</b></a>
        //! see also Eq. (88)-(96) in
        //! <a href="https://doi.org/10.1016/j.jcp.2017.05.040"><b>[ M. Geier et al. (2017), DOI:10.1016/j.jcp.2017.05  040 ]</b></a>
        //!
        ////////////////////////////////////////////////////////////////////////////////////
        // X - Dir
        vf::lbm::backwardInverseChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o1, c1o1);
        vf::lbm::backwardChimera(            mfaba, mfbba, mfcba, vvx, vx2);
        vf::lbm::backwardInverseChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c3o1, c1o3);
        vf::lbm::backwardChimera(            mfaab, mfbab, mfcab, vvx, vx2);
        vf::lbm::backwardChimera(            mfabb, mfbbb, mfcbb, vvx, vx2);
        vf::lbm::backwardChimera(            mfacb, mfbcb, mfccb, vvx, vx2);
        vf::lbm::backwardInverseChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c3o1, c1o3);
        vf::lbm::backwardChimera(            mfabc, mfbbc, mfcbc, vvx, vx2);
        vf::lbm::backwardInverseChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c9o1, c1o9);    
        ////////////////////////////////////////////////////////////////////////////////////
        // Y - Dir
        vf::lbm::backwardInverseChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2,  c6o1,  c1o6);
        vf::lbm::backwardChimera(            mfaab, mfabb, mfacb, vvy, vy2);
        vf::lbm::backwardInverseChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c18o1, c1o18);
        vf::lbm::backwardInverseChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2,  c3o2,  c2o3);
        vf::lbm::backwardChimera(            mfbab, mfbbb, mfbcb, vvy, vy2);
        vf::lbm::backwardInverseChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2,  c9o2,  c2o9);
        vf::lbm::backwardInverseChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2,  c6o1,  c1o6);
        vf::lbm::backwardChimera(            mfcab, mfcbb, mfccb, vvy, vy2);
        vf::lbm::backwardInverseChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c18o1, c1o18);  
        ////////////////////////////////////////////////////////////////////////////////////
        // Z - Dir
        vf::lbm::backwardInverseChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, c36o1, c1o36);
        vf::lbm::backwardInverseChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2,  c9o1,  c1o9);
        vf::lbm::backwardInverseChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c36o1, c1o36);
        vf::lbm::backwardInverseChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2,  c9o1,  c1o9);
        vf::lbm::backwardInverseChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2,  c9o4,  c4o9);
        vf::lbm::backwardInverseChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2,  c9o1,  c1o9);
        vf::lbm::backwardInverseChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c36o1, c1o36);
        vf::lbm::backwardInverseChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2,  c9o1,  c1o9);
        vf::lbm::backwardInverseChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c36o1, c1o36);

        ////////////////////////////////////////////////////////////////////////////////////
        //! - Write distributions: style of reading and writing the distributions from/to 
        //! stored arrays dependent on timestep is based on the esoteric twist algorithm
        //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
        //!
        (dist.f[dirE   ])[k   ] = mfabb;
        (dist.f[dirW   ])[kw  ] = mfcbb;
        (dist.f[dirN   ])[k   ] = mfbab;
        (dist.f[dirS   ])[ks  ] = mfbcb;
        (dist.f[dirT   ])[k   ] = mfbba;
        (dist.f[dirB   ])[kb  ] = mfbbc;
        (dist.f[dirNE  ])[k   ] = mfaab;
        (dist.f[dirSW  ])[ksw ] = mfccb;
        (dist.f[dirSE  ])[ks  ] = mfacb;
        (dist.f[dirNW  ])[kw  ] = mfcab;
        (dist.f[dirTE  ])[k   ] = mfaba;
        (dist.f[dirBW  ])[kbw ] = mfcbc;
        (dist.f[dirBE  ])[kb  ] = mfabc;
        (dist.f[dirTW  ])[kw  ] = mfcba;
        (dist.f[dirTN  ])[k   ] = mfbaa;
        (dist.f[dirBS  ])[kbs ] = mfbcc;
        (dist.f[dirBN  ])[kb  ] = mfbac;
        (dist.f[dirTS  ])[ks  ] = mfbca;
        (dist.f[dirREST])[k   ] = mfbbb;
        (dist.f[dirTNE ])[k   ] = mfaaa;
        (dist.f[dirTSE ])[ks  ] = mfaca;
        (dist.f[dirBNE ])[kb  ] = mfaac;
        (dist.f[dirBSE ])[kbs ] = mfacc;
        (dist.f[dirTNW ])[kw  ] = mfcaa;
        (dist.f[dirTSW ])[ksw ] = mfcca;
        (dist.f[dirBNW ])[kbw ] = mfcac;
        (dist.f[dirBSW ])[kbsw] = mfccc;
    }
}
