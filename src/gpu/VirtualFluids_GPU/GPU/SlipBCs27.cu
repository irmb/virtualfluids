#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file SlipBCs27.cu
//! \ingroup GPU
//! \author Martin Schoenherr, Anna Wellmann
//======================================================================================
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include "lbm/constants/NumericConstants.h"
#include "LBM/GPUHelperFunctions/KernelUtilities.h"

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;
using namespace vf::gpu;

//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDevice27(
    real* DD, 
    int* k_Q, 
    real* QQ,
    unsigned int numberOfBCnodes,
    real om1, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[DIR_P00] = &DD[DIR_P00 * numberOfLBnodes];
      D.f[DIR_M00] = &DD[DIR_M00 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
   } 
   else
   {
      D.f[DIR_M00] = &DD[DIR_P00 * numberOfLBnodes];
      D.f[DIR_P00] = &DD[DIR_M00 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[DIR_P00 * numberOfBCnodes];
      q_dirW   = &QQ[DIR_M00 * numberOfBCnodes];
      q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[DIR_P00])[ke   ];
      real f_E    = (D.f[DIR_M00])[kw   ];
      real f_S    = (D.f[DIR_0P0])[kn   ];
      real f_N    = (D.f[DIR_0M0])[ks   ];
      real f_B    = (D.f[DIR_00P])[kt   ];
      real f_T    = (D.f[DIR_00M])[kb   ];
      real f_SW   = (D.f[DIR_PP0])[kne  ];
      real f_NE   = (D.f[DIR_MM0])[ksw  ];
      real f_NW   = (D.f[DIR_PM0])[kse  ];
      real f_SE   = (D.f[DIR_MP0])[knw  ];
      real f_BW   = (D.f[DIR_P0P])[kte  ];
      real f_TE   = (D.f[DIR_M0M])[kbw  ];
      real f_TW   = (D.f[DIR_P0M])[kbe  ];
      real f_BE   = (D.f[DIR_M0P])[ktw  ];
      real f_BS   = (D.f[DIR_0PP])[ktn  ];
      real f_TN   = (D.f[DIR_0MM])[kbs  ];
      real f_TS   = (D.f[DIR_0PM])[kbn  ];
      real f_BN   = (D.f[DIR_0MP])[kts  ];
      real f_BSW  = (D.f[DIR_PPP])[ktne ];
      real f_BNE  = (D.f[DIR_MMP])[ktsw ];
      real f_BNW  = (D.f[DIR_PMP])[ktse ];
      real f_BSE  = (D.f[DIR_MPP])[ktnw ];
      real f_TSW  = (D.f[DIR_PPM])[kbne ];
      real f_TNE  = (D.f[DIR_MMM])[kbsw ];
      real f_TNW  = (D.f[DIR_PMM])[kbse ];
      real f_TSE  = (D.f[DIR_MPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[DIR_000])[kzero]); 

      vx1    =  ((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W); 
         

      vx2    =   (-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S); 

      vx3    =   ((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[DIR_P00] = &DD[DIR_P00 * numberOfLBnodes];
         D.f[DIR_M00] = &DD[DIR_M00 * numberOfLBnodes];
         D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
         D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
         D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
         D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
         D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
         D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
         D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
         D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
         D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
         D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
         D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
         D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
         D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
         D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
         D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
         D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
         D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
         D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
         D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
         D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
         D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
         D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
         D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
         D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
      } 
      else
      {
         D.f[DIR_M00] = &DD[DIR_P00 * numberOfLBnodes];
         D.f[DIR_P00] = &DD[DIR_M00 * numberOfLBnodes];
         D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
         D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
         D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
         D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
         D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
         D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
         D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
         D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
         D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
         D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
         D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
         D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
         D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
         D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
         D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
         D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
         D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
         D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
         D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
         D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
         D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
         D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
         D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
         D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[DIR_000])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real fac = c1o1;//c99o100;
	  real VeloX = fac*vx1;
	  real VeloY = fac*vx2;
	  real VeloZ = fac*vx3;
	  bool x = false;
	  bool y = false;
	  bool z = false;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho+c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq); 
         (D.f[DIR_M00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-feq*om1)/(c1o1-om1)+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q);
         //(D.f[DIR_M00])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho+c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq); 
         (D.f[DIR_P00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-feq*om1)/(c1o1-om1)+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q);
         //(D.f[DIR_P00])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho+c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq); 
         (D.f[DIR_0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-feq*om1)/(c1o1-om1)+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q);
         //(D.f[DIR_0M0])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho+c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq); 
         (D.f[DIR_0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-feq*om1)/(c1o1-om1)+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q);
         //(D.f[DIR_0P0])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho+c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq); 
         (D.f[DIR_00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-feq*om1)/(c1o1-om1)+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q);
         //(D.f[DIR_00M])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho+c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq); 
         (D.f[DIR_00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-feq*om1)/(c1o1-om1)+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q);
         //(D.f[DIR_00P])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         (D.f[DIR_MM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-feq*om1)/(c1o1-om1)+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
         //(D.f[DIR_MM0])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         (D.f[DIR_PP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-feq*om1)/(c1o1-om1)+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
         //(D.f[DIR_PP0])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         (D.f[DIR_MP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-feq*om1)/(c1o1-om1)+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
         //(D.f[DIR_MP0])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         (D.f[DIR_PM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-feq*om1)/(c1o1-om1)+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
         //(D.f[DIR_PM0])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         (D.f[DIR_M0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-feq*om1)/(c1o1-om1)+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
         //(D.f[DIR_M0M])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         (D.f[DIR_P0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-feq*om1)/(c1o1-om1)+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
         //(D.f[DIR_P0P])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         (D.f[DIR_M0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-feq*om1)/(c1o1-om1)+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
         //(D.f[DIR_M0P])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         (D.f[DIR_P0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-feq*om1)/(c1o1-om1)+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
         //(D.f[DIR_P0M])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         (D.f[DIR_0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-feq*om1)/(c1o1-om1)+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
         //(D.f[DIR_0MM])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         (D.f[DIR_0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-feq*om1)/(c1o1-om1)+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
         //(D.f[DIR_0PP])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         (D.f[DIR_0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-feq*om1)/(c1o1-om1)+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
         //(D.f[DIR_0MP])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         (D.f[DIR_0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-feq*om1)/(c1o1-om1)+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
         //(D.f[DIR_0PM])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         (D.f[DIR_MMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-feq*om1)/(c1o1-om1)+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
         //(D.f[DIR_MMM])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         (D.f[DIR_PPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-feq*om1)/(c1o1-om1)+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
         //(D.f[DIR_PPP])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         (D.f[DIR_MMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-feq*om1)/(c1o1-om1)+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
         //(D.f[DIR_MMP])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         (D.f[DIR_PPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-feq*om1)/(c1o1-om1)+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
         //(D.f[DIR_PPM])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         (D.f[DIR_MPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-feq*om1)/(c1o1-om1)+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
         //(D.f[DIR_MPM])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         (D.f[DIR_PMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-feq*om1)/(c1o1-om1)+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
         //(D.f[DIR_PMP])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         (D.f[DIR_MPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-feq*om1)/(c1o1-om1)+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
         //(D.f[DIR_MPP])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         (D.f[DIR_PMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-feq*om1)/(c1o1-om1)+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
         //(D.f[DIR_PMM])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDeviceComp27(
    real* distributions, 
    int* subgridDistanceIndices, 
    real* subgridDistances,
    unsigned int numberOfBCnodes,
    real omega, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!

   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned nodeIndex = getNodeIndex();

   if(nodeIndex < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[nodeIndex];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[DIR_P00])[ke   ];
      real f_E    = (dist.f[DIR_M00])[kw   ];
      real f_S    = (dist.f[DIR_0P0])[kn   ];
      real f_N    = (dist.f[DIR_0M0])[ks   ];
      real f_B    = (dist.f[DIR_00P])[kt   ];
      real f_T    = (dist.f[DIR_00M])[kb   ];
      real f_SW   = (dist.f[DIR_PP0])[kne  ];
      real f_NE   = (dist.f[DIR_MM0])[ksw  ];
      real f_NW   = (dist.f[DIR_PM0])[kse  ];
      real f_SE   = (dist.f[DIR_MP0])[knw  ];
      real f_BW   = (dist.f[DIR_P0P])[kte  ];
      real f_TE   = (dist.f[DIR_M0M])[kbw  ];
      real f_TW   = (dist.f[DIR_P0M])[kbe  ];
      real f_BE   = (dist.f[DIR_M0P])[ktw  ];
      real f_BS   = (dist.f[DIR_0PP])[ktn  ];
      real f_TN   = (dist.f[DIR_0MM])[kbs  ];
      real f_TS   = (dist.f[DIR_0PM])[kbn  ];
      real f_BN   = (dist.f[DIR_0MP])[kts  ];
      real f_BSW  = (dist.f[DIR_PPP])[ktne ];
      real f_BNE  = (dist.f[DIR_MMP])[ktsw ];
      real f_BNW  = (dist.f[DIR_PMP])[ktse ];
      real f_BSE  = (dist.f[DIR_MPP])[ktnw ];
      real f_TSW  = (dist.f[DIR_PPM])[kbne ];
      real f_TNE  = (dist.f[DIR_MMM])[kbsw ];
      real f_TNW  = (dist.f[DIR_PMM])[kbse ];
      real f_TSE  = (dist.f[DIR_MPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[DIR_000])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real feq, q, velocityLB, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[DIR_P00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloX;
         (dist.f[DIR_M00])[kw] = getInterpolatedDistributionForVeloBC(q, f_E, f_W, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_M00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloX;
         (dist.f[DIR_P00])[ke] = getInterpolatedDistributionForVeloBC(q, f_W, f_E, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_0P0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloY;
         (dist.f[DIR_0M0])[ks] = getInterpolatedDistributionForVeloBC(q, f_N, f_S, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_0M0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloY;
         (dist.f[DIR_0P0])[kn] = getInterpolatedDistributionForVeloBC(q, f_S, f_N, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_00P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloZ;
         (dist.f[DIR_00M])[kb] = getInterpolatedDistributionForVeloBC(q, f_T, f_B, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_00M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloZ;
         (dist.f[DIR_00P])[kt] = getInterpolatedDistributionForVeloBC(q, f_B, f_T, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_PP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloY;
         (dist.f[DIR_MM0])[ksw] = getInterpolatedDistributionForVeloBC(q, f_NE, f_SW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_MM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloY;
         (dist.f[DIR_PP0])[kne] = getInterpolatedDistributionForVeloBC(q, f_SW, f_NE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_PM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloY;
         (dist.f[DIR_MP0])[knw] = getInterpolatedDistributionForVeloBC(q, f_SE, f_NW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_MP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloY;
         (dist.f[DIR_PM0])[kse] = getInterpolatedDistributionForVeloBC(q, f_NW, f_SE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_P0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloZ;
         (dist.f[DIR_M0M])[kbw] = getInterpolatedDistributionForVeloBC(q, f_TE, f_BW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_M0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloZ;
         (dist.f[DIR_P0P])[kte] = getInterpolatedDistributionForVeloBC(q, f_BW, f_TE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_P0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloZ;
         (dist.f[DIR_M0P])[ktw] = getInterpolatedDistributionForVeloBC(q, f_BE, f_TW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_M0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloZ;
         (dist.f[DIR_P0M])[kbe] = getInterpolatedDistributionForVeloBC(q, f_TW, f_BE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0PP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY + VeloZ;
         (dist.f[DIR_0MM])[kbs] = getInterpolatedDistributionForVeloBC(q, f_TN, f_BS, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0MM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY - VeloZ;
         (dist.f[DIR_0PP])[ktn] = getInterpolatedDistributionForVeloBC(q, f_BS, f_TN, feq, omega, velocityBC, c1o54);
      }


      q = (subgridD.q[DIR_0PM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY - VeloZ;
         (dist.f[DIR_0MP])[kts] = getInterpolatedDistributionForVeloBC(q, f_BN, f_TS, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0MP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY + VeloZ;
         (dist.f[DIR_0PM])[kbn] = getInterpolatedDistributionForVeloBC(q, f_TS, f_BN, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_PPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[DIR_MMM])[kbsw] = getInterpolatedDistributionForVeloBC(q, f_TNE, f_BSW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[DIR_PPP])[ktne] = getInterpolatedDistributionForVeloBC(q, f_BSW, f_TNE, feq, omega, velocityBC, c1o216);
      }


      q = (subgridD.q[DIR_PPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[DIR_MMP])[ktsw] = getInterpolatedDistributionForVeloBC(q, f_BNE, f_TSW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[DIR_PPM])[kbne] = getInterpolatedDistributionForVeloBC(q, f_TSW, f_BNE, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_PMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[DIR_MPM])[kbnw] = getInterpolatedDistributionForVeloBC(q, f_TSE, f_BNW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[DIR_PMP])[ktse] = getInterpolatedDistributionForVeloBC(q, f_BNW, f_TSE, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_PMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[DIR_MPP])[ktnw] = getInterpolatedDistributionForVeloBC(q, f_BSE, f_TNW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[DIR_PMM])[kbse] = getInterpolatedDistributionForVeloBC(q, f_TNW, f_BSE, feq, omega, velocityBC, c1o216);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



























//////////////////////////////////////////////////////////////////////////////
__global__ void BBSlipDeviceComp27(
    real* distributions, 
    int* subgridDistanceIndices, 
    real* subgridDistances,
    unsigned int numberOfBCnodes,
    real omega, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!

   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned nodeIndex = getNodeIndex();

   if(nodeIndex < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[nodeIndex];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[DIR_P00])[ke   ];
      real f_E    = (dist.f[DIR_M00])[kw   ];
      real f_S    = (dist.f[DIR_0P0])[kn   ];
      real f_N    = (dist.f[DIR_0M0])[ks   ];
      real f_B    = (dist.f[DIR_00P])[kt   ];
      real f_T    = (dist.f[DIR_00M])[kb   ];
      real f_SW   = (dist.f[DIR_PP0])[kne  ];
      real f_NE   = (dist.f[DIR_MM0])[ksw  ];
      real f_NW   = (dist.f[DIR_PM0])[kse  ];
      real f_SE   = (dist.f[DIR_MP0])[knw  ];
      real f_BW   = (dist.f[DIR_P0P])[kte  ];
      real f_TE   = (dist.f[DIR_M0M])[kbw  ];
      real f_TW   = (dist.f[DIR_P0M])[kbe  ];
      real f_BE   = (dist.f[DIR_M0P])[ktw  ];
      real f_BS   = (dist.f[DIR_0PP])[ktn  ];
      real f_TN   = (dist.f[DIR_0MM])[kbs  ];
      real f_TS   = (dist.f[DIR_0PM])[kbn  ];
      real f_BN   = (dist.f[DIR_0MP])[kts  ];
      real f_BSW  = (dist.f[DIR_PPP])[ktne ];
      real f_BNE  = (dist.f[DIR_MMP])[ktsw ];
      real f_BNW  = (dist.f[DIR_PMP])[ktse ];
      real f_BSE  = (dist.f[DIR_MPP])[ktnw ];
      real f_TSW  = (dist.f[DIR_PPM])[kbne ];
      real f_TNE  = (dist.f[DIR_MMM])[kbsw ];
      real f_TNW  = (dist.f[DIR_PMM])[kbse ];
      real f_TSE  = (dist.f[DIR_MPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[DIR_000])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real q, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[DIR_P00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityBC = VeloX;
         (dist.f[DIR_M00])[kw] = getBounceBackDistributionForVeloBC(f_W, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_M00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityBC = -VeloX;
         (dist.f[DIR_P00])[ke] = getBounceBackDistributionForVeloBC(f_E, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_0P0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityBC = VeloY;
         (dist.f[DIR_0M0])[ks] = getBounceBackDistributionForVeloBC(f_S, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_0M0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityBC = -VeloY;
         (dist.f[DIR_0P0])[kn] = getBounceBackDistributionForVeloBC(f_N, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_00P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityBC = VeloZ;
         (dist.f[DIR_00M])[kb] = getBounceBackDistributionForVeloBC(f_B, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_00M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityBC = -VeloZ;
         (dist.f[DIR_00P])[kt] = getBounceBackDistributionForVeloBC(f_T, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_PP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityBC = VeloX + VeloY;
         (dist.f[DIR_MM0])[ksw] = getBounceBackDistributionForVeloBC(f_SW, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_MM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityBC = -VeloX - VeloY;
         (dist.f[DIR_PP0])[kne] = getBounceBackDistributionForVeloBC(f_NE, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_PM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityBC = VeloX - VeloY;
         (dist.f[DIR_MP0])[knw] = getBounceBackDistributionForVeloBC(f_NW, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_MP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityBC = -VeloX + VeloY;
         (dist.f[DIR_PM0])[kse] = getBounceBackDistributionForVeloBC(f_SE, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_P0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX + VeloZ;
         (dist.f[DIR_M0M])[kbw] = getBounceBackDistributionForVeloBC(f_BW, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_M0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

        velocityBC = -VeloX - VeloZ;
        (dist.f[DIR_P0P])[kte] = getBounceBackDistributionForVeloBC(f_TE, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_P0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX - VeloZ;
         (dist.f[DIR_M0P])[ktw] = getBounceBackDistributionForVeloBC(f_TW, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_M0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX + VeloZ;
         (dist.f[DIR_P0M])[kbe] = getBounceBackDistributionForVeloBC(f_BE, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0PP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloY + VeloZ;
         (dist.f[DIR_0MM])[kbs] = getBounceBackDistributionForVeloBC(f_BS, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0MM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloY - VeloZ;
         (dist.f[DIR_0PP])[ktn] = getBounceBackDistributionForVeloBC(f_TN, velocityBC, c1o54);
      }


      q = (subgridD.q[DIR_0PM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloY - VeloZ;
         (dist.f[DIR_0MP])[kts] = getBounceBackDistributionForVeloBC(f_TS, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0MP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloY + VeloZ;
         (dist.f[DIR_0PM])[kbn] = getBounceBackDistributionForVeloBC(f_BN, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_PPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[DIR_MMM])[kbsw] = getBounceBackDistributionForVeloBC(f_TNE, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[DIR_PPP])[ktne] = getBounceBackDistributionForVeloBC(f_TNE, velocityBC, c1o216);
      }


      q = (subgridD.q[DIR_PPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[DIR_MMP])[ktsw] = getBounceBackDistributionForVeloBC(f_TSW, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[DIR_PPM])[kbne] = getBounceBackDistributionForVeloBC(f_BNE, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_PMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[DIR_MPM])[kbnw] = getBounceBackDistributionForVeloBC(f_BNW, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[DIR_PMP])[ktse] = getBounceBackDistributionForVeloBC(f_TSE, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_PMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[DIR_MPP])[ktnw] = getBounceBackDistributionForVeloBC(f_TNW, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[DIR_PMM])[kbse] = getBounceBackDistributionForVeloBC(f_BSE, velocityBC, c1o216);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////




























////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDeviceComp27TurbViscosity(
    real* distributions, 
    int* subgridDistanceIndices, 
    real* subgridDistances,
    unsigned int numberOfBCnodes,
    real omega, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    real* turbViscosity,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!

   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned nodeIndex = getNodeIndex();

   if(nodeIndex < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[nodeIndex];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[DIR_P00])[ke   ];
      real f_E    = (dist.f[DIR_M00])[kw   ];
      real f_S    = (dist.f[DIR_0P0])[kn   ];
      real f_N    = (dist.f[DIR_0M0])[ks   ];
      real f_B    = (dist.f[DIR_00P])[kt   ];
      real f_T    = (dist.f[DIR_00M])[kb   ];
      real f_SW   = (dist.f[DIR_PP0])[kne  ];
      real f_NE   = (dist.f[DIR_MM0])[ksw  ];
      real f_NW   = (dist.f[DIR_PM0])[kse  ];
      real f_SE   = (dist.f[DIR_MP0])[knw  ];
      real f_BW   = (dist.f[DIR_P0P])[kte  ];
      real f_TE   = (dist.f[DIR_M0M])[kbw  ];
      real f_TW   = (dist.f[DIR_P0M])[kbe  ];
      real f_BE   = (dist.f[DIR_M0P])[ktw  ];
      real f_BS   = (dist.f[DIR_0PP])[ktn  ];
      real f_TN   = (dist.f[DIR_0MM])[kbs  ];
      real f_TS   = (dist.f[DIR_0PM])[kbn  ];
      real f_BN   = (dist.f[DIR_0MP])[kts  ];
      real f_BSW  = (dist.f[DIR_PPP])[ktne ];
      real f_BNE  = (dist.f[DIR_MMP])[ktsw ];
      real f_BNW  = (dist.f[DIR_PMP])[ktse ];
      real f_BSE  = (dist.f[DIR_MPP])[ktnw ];
      real f_TSW  = (dist.f[DIR_PPM])[kbne ];
      real f_TNE  = (dist.f[DIR_MMM])[kbsw ];
      real f_TNW  = (dist.f[DIR_PMM])[kbse ];
      real f_TSE  = (dist.f[DIR_MPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[DIR_000])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - compute local relaxation rate
      //!
      real om_turb = omega / (c1o1 + c3o1* omega* max(c0o1, turbViscosity[indexOfBCnode]) );

      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real feq, q, velocityLB, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[DIR_P00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloX;
         (dist.f[DIR_M00])[kw] = getInterpolatedDistributionForVeloBC(q, f_E, f_W, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_M00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloX;
         (dist.f[DIR_P00])[ke] = getInterpolatedDistributionForVeloBC(q, f_W, f_E, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_0P0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloY;
         (dist.f[DIR_0M0])[ks] = getInterpolatedDistributionForVeloBC(q, f_N, f_S, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_0M0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloY;
         (dist.f[DIR_0P0])[kn] = getInterpolatedDistributionForVeloBC(q, f_S, f_N, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_00P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloZ;
         (dist.f[DIR_00M])[kb] = getInterpolatedDistributionForVeloBC(q, f_T, f_B, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_00M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloZ;
         (dist.f[DIR_00P])[kt] = getInterpolatedDistributionForVeloBC(q, f_B, f_T, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_PP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloY;
         (dist.f[DIR_MM0])[ksw] = getInterpolatedDistributionForVeloBC(q, f_NE, f_SW, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_MM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloY;
         (dist.f[DIR_PP0])[kne] = getInterpolatedDistributionForVeloBC(q, f_SW, f_NE, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_PM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloY;
         (dist.f[DIR_MP0])[knw] = getInterpolatedDistributionForVeloBC(q, f_SE, f_NW, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_MP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloY;
         (dist.f[DIR_PM0])[kse] = getInterpolatedDistributionForVeloBC(q, f_NW, f_SE, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_P0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloZ;
         (dist.f[DIR_M0M])[kbw] = getInterpolatedDistributionForVeloBC(q, f_TE, f_BW, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_M0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

        velocityLB = -vx1 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloX - VeloZ;
        (dist.f[DIR_P0P])[kte] = getInterpolatedDistributionForVeloBC(q, f_BW, f_TE, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_P0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloZ;
         (dist.f[DIR_M0P])[ktw] = getInterpolatedDistributionForVeloBC(q, f_BE, f_TW, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_M0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloZ;
         (dist.f[DIR_P0M])[kbe] = getInterpolatedDistributionForVeloBC(q, f_TW, f_BE, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0PP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY + VeloZ;
         (dist.f[DIR_0MM])[kbs] = getInterpolatedDistributionForVeloBC(q, f_TN, f_BS, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0MM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY - VeloZ;
         (dist.f[DIR_0PP])[ktn] = getInterpolatedDistributionForVeloBC(q, f_BS, f_TN, feq, om_turb, velocityBC, c1o54);
      }


      q = (subgridD.q[DIR_0PM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY - VeloZ;
         (dist.f[DIR_0MP])[kts] = getInterpolatedDistributionForVeloBC(q, f_BN, f_TS, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0MP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY + VeloZ;
         (dist.f[DIR_0PM])[kbn] = getInterpolatedDistributionForVeloBC(q, f_TS, f_BN, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_PPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[DIR_MMM])[kbsw] = getInterpolatedDistributionForVeloBC(q, f_TNE, f_BSW, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[DIR_PPP])[ktne] = getInterpolatedDistributionForVeloBC(q, f_BSW, f_TNE, feq, om_turb, velocityBC, c1o216);
      }


      q = (subgridD.q[DIR_PPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[DIR_MMP])[ktsw] = getInterpolatedDistributionForVeloBC(q, f_BNE, f_TSW, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[DIR_PPM])[kbne] = getInterpolatedDistributionForVeloBC(q, f_TSW, f_BNE, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_PMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[DIR_MPM])[kbnw] = getInterpolatedDistributionForVeloBC(q, f_TSE, f_BNW, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[DIR_PMP])[ktse] = getInterpolatedDistributionForVeloBC(q, f_BNW, f_TSE, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_PMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[DIR_MPP])[ktnw] = getInterpolatedDistributionForVeloBC(q, f_BSE, f_TNW, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[DIR_PMM])[kbse] = getInterpolatedDistributionForVeloBC(q, f_TNW, f_BSE, feq, om_turb, velocityBC, c1o216);
      }
   }
}
////////////////////////////////////////////////////////////////////////////






























////////////////////////////////////////////////////////////////////////////
__global__ void QSlipPressureDeviceComp27TurbViscosity(
    real* distributions, 
    int* subgridDistanceIndices, 
    real* subgridDistances,
    unsigned int numberOfBCnodes,
    real omega, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    real* turbViscosity,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!
   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned nodeIndex = getNodeIndex();

   if(nodeIndex < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[nodeIndex];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[DIR_P00])[ke   ];
      real f_E    = (dist.f[DIR_M00])[kw   ];
      real f_S    = (dist.f[DIR_0P0])[kn   ];
      real f_N    = (dist.f[DIR_0M0])[ks   ];
      real f_B    = (dist.f[DIR_00P])[kt   ];
      real f_T    = (dist.f[DIR_00M])[kb   ];
      real f_SW   = (dist.f[DIR_PP0])[kne  ];
      real f_NE   = (dist.f[DIR_MM0])[ksw  ];
      real f_NW   = (dist.f[DIR_PM0])[kse  ];
      real f_SE   = (dist.f[DIR_MP0])[knw  ];
      real f_BW   = (dist.f[DIR_P0P])[kte  ];
      real f_TE   = (dist.f[DIR_M0M])[kbw  ];
      real f_TW   = (dist.f[DIR_P0M])[kbe  ];
      real f_BE   = (dist.f[DIR_M0P])[ktw  ];
      real f_BS   = (dist.f[DIR_0PP])[ktn  ];
      real f_TN   = (dist.f[DIR_0MM])[kbs  ];
      real f_TS   = (dist.f[DIR_0PM])[kbn  ];
      real f_BN   = (dist.f[DIR_0MP])[kts  ];
      real f_BSW  = (dist.f[DIR_PPP])[ktne ];
      real f_BNE  = (dist.f[DIR_MMP])[ktsw ];
      real f_BNW  = (dist.f[DIR_PMP])[ktse ];
      real f_BSE  = (dist.f[DIR_MPP])[ktnw ];
      real f_TSW  = (dist.f[DIR_PPM])[kbne ];
      real f_TNE  = (dist.f[DIR_MMM])[kbsw ];
      real f_TNW  = (dist.f[DIR_PMM])[kbse ];
      real f_TSE  = (dist.f[DIR_MPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[DIR_000])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - compute local relaxation rate
      //!
      real om_turb = omega / (c1o1 + c3o1* omega* max(c0o1, turbViscosity[indexOfBCnode]) );

      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real feq, q, velocityLB, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[DIR_P00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloX;
         (dist.f[DIR_M00])[kw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_E, f_W, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_M00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloX;
         (dist.f[DIR_P00])[ke] = getInterpolatedDistributionForVeloWithPressureBC(q, f_W, f_E, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_0P0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloY;
         (dist.f[DIR_0M0])[ks] = getInterpolatedDistributionForVeloWithPressureBC(q, f_N, f_S, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_0M0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloY;
         (dist.f[DIR_0P0])[kn] = getInterpolatedDistributionForVeloWithPressureBC(q, f_S, f_N, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_00P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloZ;
         (dist.f[DIR_00M])[kb] = getInterpolatedDistributionForVeloWithPressureBC(q, f_T, f_B, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_00M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloZ;
         (dist.f[DIR_00P])[kt] = getInterpolatedDistributionForVeloWithPressureBC(q, f_B, f_T, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[DIR_PP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloY;
         (dist.f[DIR_MM0])[ksw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_NE, f_SW, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_MM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloY;
         (dist.f[DIR_PP0])[kne] = getInterpolatedDistributionForVeloWithPressureBC(q, f_SW, f_NE, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_PM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloY;
         (dist.f[DIR_MP0])[knw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_SE, f_NW, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_MP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloY;
         (dist.f[DIR_PM0])[kse] = getInterpolatedDistributionForVeloWithPressureBC(q, f_NW, f_SE, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_P0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloZ;
         (dist.f[DIR_M0M])[kbw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TE, f_BW, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_M0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

        velocityLB = -vx1 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloX - VeloZ;
        (dist.f[DIR_P0P])[kte] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BW, f_TE, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_P0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloZ;
         (dist.f[DIR_M0P])[ktw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BE, f_TW, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_M0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloZ;
         (dist.f[DIR_P0M])[kbe] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TW, f_BE, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0PP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY + VeloZ;
         (dist.f[DIR_0MM])[kbs] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TN, f_BS, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0MM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY - VeloZ;
         (dist.f[DIR_0PP])[ktn] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BS, f_TN, feq, om_turb, drho, velocityBC, c1o54);
      }


      q = (subgridD.q[DIR_0PM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY - VeloZ;
         (dist.f[DIR_0MP])[kts] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BN, f_TS, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_0MP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY + VeloZ;
         (dist.f[DIR_0PM])[kbn] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TS, f_BN, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[DIR_PPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[DIR_MMM])[kbsw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TNE, f_BSW, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[DIR_PPP])[ktne] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BSW, f_TNE, feq, om_turb, drho, velocityBC, c1o216);
      }


      q = (subgridD.q[DIR_PPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[DIR_MMP])[ktsw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BNE, f_TSW, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[DIR_PPM])[kbne] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TSW, f_BNE, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_PMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[DIR_MPM])[kbnw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TSE, f_BNW, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[DIR_PMP])[ktse] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BNW, f_TSE, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_PMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[DIR_MPP])[ktnw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BSE, f_TNW, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[DIR_MPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[DIR_PMM])[kbse] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TNW, f_BSE, feq, om_turb, drho, velocityBC, c1o216);
      }
   }
}

// __global__ void QSlipDeviceComp27TurbViscosity(real* DD, 
// 											 int* k_Q, 
// 											 real* QQ,
// 											 unsigned int numberOfBCnodes,
// 											 real om1, 
// 											 unsigned int* neighborX,
// 											 unsigned int* neighborY,
// 											 unsigned int* neighborZ,
//                                   real* turbViscosity,
// 											 unsigned int size_Mat, 
// 											 bool isEvenTimestep)
// {
//    Distributions27 D;
//    if (isEvenTimestep==true)
//    {
//       D.f[DIR_P00] = &DD[DIR_P00 * size_Mat];
//       D.f[DIR_M00] = &DD[DIR_M00 * size_Mat];
//       D.f[DIR_0P0] = &DD[DIR_0P0 * size_Mat];
//       D.f[DIR_0M0] = &DD[DIR_0M0 * size_Mat];
//       D.f[DIR_00P] = &DD[DIR_00P * size_Mat];
//       D.f[DIR_00M] = &DD[DIR_00M * size_Mat];
//       D.f[DIR_PP0] = &DD[DIR_PP0 * size_Mat];
//       D.f[DIR_MM0] = &DD[DIR_MM0 * size_Mat];
//       D.f[DIR_PM0] = &DD[DIR_PM0 * size_Mat];
//       D.f[DIR_MP0] = &DD[DIR_MP0 * size_Mat];
//       D.f[DIR_P0P] = &DD[DIR_P0P * size_Mat];
//       D.f[DIR_M0M] = &DD[DIR_M0M * size_Mat];
//       D.f[DIR_P0M] = &DD[DIR_P0M * size_Mat];
//       D.f[DIR_M0P] = &DD[DIR_M0P * size_Mat];
//       D.f[DIR_0PP] = &DD[DIR_0PP * size_Mat];
//       D.f[DIR_0MM] = &DD[DIR_0MM * size_Mat];
//       D.f[DIR_0PM] = &DD[DIR_0PM * size_Mat];
//       D.f[DIR_0MP] = &DD[DIR_0MP * size_Mat];
//       D.f[DIR_000] = &DD[DIR_000 * size_Mat];
//       D.f[DIR_PPP] = &DD[DIR_PPP * size_Mat];
//       D.f[DIR_MMP] = &DD[DIR_MMP * size_Mat];
//       D.f[DIR_PMP] = &DD[DIR_PMP * size_Mat];
//       D.f[DIR_MPP] = &DD[DIR_MPP * size_Mat];
//       D.f[DIR_PPM] = &DD[DIR_PPM * size_Mat];
//       D.f[DIR_MMM] = &DD[DIR_MMM * size_Mat];
//       D.f[DIR_PMM] = &DD[DIR_PMM * size_Mat];
//       D.f[DIR_MPM] = &DD[DIR_MPM * size_Mat];
//    } 
//    else
//    {
//       D.f[DIR_M00] = &DD[DIR_P00 * size_Mat];
//       D.f[DIR_P00] = &DD[DIR_M00 * size_Mat];
//       D.f[DIR_0M0] = &DD[DIR_0P0 * size_Mat];
//       D.f[DIR_0P0] = &DD[DIR_0M0 * size_Mat];
//       D.f[DIR_00M] = &DD[DIR_00P * size_Mat];
//       D.f[DIR_00P] = &DD[DIR_00M * size_Mat];
//       D.f[DIR_MM0] = &DD[DIR_PP0 * size_Mat];
//       D.f[DIR_PP0] = &DD[DIR_MM0 * size_Mat];
//       D.f[DIR_MP0] = &DD[DIR_PM0 * size_Mat];
//       D.f[DIR_PM0] = &DD[DIR_MP0 * size_Mat];
//       D.f[DIR_M0M] = &DD[DIR_P0P * size_Mat];
//       D.f[DIR_P0P] = &DD[DIR_M0M * size_Mat];
//       D.f[DIR_M0P] = &DD[DIR_P0M * size_Mat];
//       D.f[DIR_P0M] = &DD[DIR_M0P * size_Mat];
//       D.f[DIR_0MM] = &DD[DIR_0PP * size_Mat];
//       D.f[DIR_0PP] = &DD[DIR_0MM * size_Mat];
//       D.f[DIR_0MP] = &DD[DIR_0PM * size_Mat];
//       D.f[DIR_0PM] = &DD[DIR_0MP * size_Mat];
//       D.f[DIR_000] = &DD[DIR_000 * size_Mat];
//       D.f[DIR_PPP] = &DD[DIR_MMM * size_Mat];
//       D.f[DIR_MMP] = &DD[DIR_PPM * size_Mat];
//       D.f[DIR_PMP] = &DD[DIR_MPM * size_Mat];
//       D.f[DIR_MPP] = &DD[DIR_PMM * size_Mat];
//       D.f[DIR_PPM] = &DD[DIR_MMP * size_Mat];
//       D.f[DIR_MMM] = &DD[DIR_PPP * size_Mat];
//       D.f[DIR_PMM] = &DD[DIR_MPP * size_Mat];
//       D.f[DIR_MPM] = &DD[DIR_PMP * size_Mat];
//    }
//    ////////////////////////////////////////////////////////////////////////////////
//    const unsigned  x = threadIdx.x;  // Globaler x-Index 
//    const unsigned  y = blockIdx.x;   // Globaler y-Index 
//    const unsigned  z = blockIdx.y;   // Globaler z-Index 

//    const unsigned nx = blockDim.x;
//    const unsigned ny = gridDim.x;

//    const unsigned k = nx*(ny*z + y) + x;
//    //////////////////////////////////////////////////////////////////////////

//    if(k<numberOfBCnodes)
//    {
//       ////////////////////////////////////////////////////////////////////////////////
//       real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
//             *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
//             *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
//             *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
//             *q_dirBSE, *q_dirBNW; 
//       q_dirE   = &QQ[DIR_P00 * numberOfBCnodes];
//       q_dirW   = &QQ[DIR_M00 * numberOfBCnodes];
//       q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
//       q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
//       q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
//       q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
//       q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
//       q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
//       q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
//       q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
//       q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
//       q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
//       q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
//       q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
//       q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
//       q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
//       q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
//       q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
//       q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
//       q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
//       q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
//       q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
//       q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
//       q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
//       q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
//       q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
//       ////////////////////////////////////////////////////////////////////////////////
//       //index
//       unsigned int KQK  = k_Q[k];
//       unsigned int kzero= KQK;
//       unsigned int ke   = KQK;
//       unsigned int kw   = neighborX[KQK];
//       unsigned int kn   = KQK;
//       unsigned int ks   = neighborY[KQK];
//       unsigned int kt   = KQK;
//       unsigned int kb   = neighborZ[KQK];
//       unsigned int ksw  = neighborY[kw];
//       unsigned int kne  = KQK;
//       unsigned int kse  = ks;
//       unsigned int knw  = kw;
//       unsigned int kbw  = neighborZ[kw];
//       unsigned int kte  = KQK;
//       unsigned int kbe  = kb;
//       unsigned int ktw  = kw;
//       unsigned int kbs  = neighborZ[ks];
//       unsigned int ktn  = KQK;
//       unsigned int kbn  = kb;
//       unsigned int kts  = ks;
//       unsigned int ktse = ks;
//       unsigned int kbnw = kbw;
//       unsigned int ktnw = kw;
//       unsigned int kbse = kbs;
//       unsigned int ktsw = ksw;
//       unsigned int kbne = kb;
//       unsigned int ktne = KQK;
//       unsigned int kbsw = neighborZ[ksw];
      
//       ////////////////////////////////////////////////////////////////////////////////
//       real f_W    = (D.f[DIR_P00])[ke   ];
//       real f_E    = (D.f[DIR_M00])[kw   ];
//       real f_S    = (D.f[DIR_0P0])[kn   ];
//       real f_N    = (D.f[DIR_0M0])[ks   ];
//       real f_B    = (D.f[DIR_00P])[kt   ];
//       real f_T    = (D.f[DIR_00M])[kb   ];
//       real f_SW   = (D.f[DIR_PP0])[kne  ];
//       real f_NE   = (D.f[DIR_MM0])[ksw  ];
//       real f_NW   = (D.f[DIR_PM0])[kse  ];
//       real f_SE   = (D.f[DIR_MP0])[knw  ];
//       real f_BW   = (D.f[DIR_P0P])[kte  ];
//       real f_TE   = (D.f[DIR_M0M])[kbw  ];
//       real f_TW   = (D.f[DIR_P0M])[kbe  ];
//       real f_BE   = (D.f[DIR_M0P])[ktw  ];
//       real f_BS   = (D.f[DIR_0PP])[ktn  ];
//       real f_TN   = (D.f[DIR_0MM])[kbs  ];
//       real f_TS   = (D.f[DIR_0PM])[kbn  ];
//       real f_BN   = (D.f[DIR_0MP])[kts  ];
//       real f_BSW  = (D.f[DIR_PPP])[ktne ];
//       real f_BNE  = (D.f[DIR_MMP])[ktsw ];
//       real f_BNW  = (D.f[DIR_PMP])[ktse ];
//       real f_BSE  = (D.f[DIR_MPP])[ktnw ];
//       real f_TSW  = (D.f[DIR_PPM])[kbne ];
//       real f_TNE  = (D.f[DIR_MMM])[kbsw ];
//       real f_TNW  = (D.f[DIR_PMM])[kbse ];
//       real f_TSE  = (D.f[DIR_MPM])[kbnw ];
//       ////////////////////////////////////////////////////////////////////////////////
//       real vx1, vx2, vx3, drho, feq, q;
//       drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
//                 f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
//                 f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[DIR_000])[kzero]); 

//       vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
//                 ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
//                 (f_E - f_W)) / (c1o1 + drho); 
         

//       vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
//                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
//                  (f_N - f_S)) / (c1o1 + drho); 

//       vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
//                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
//                  (f_T - f_B)) / (c1o1 + drho); 

//       real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

//       //////////////////////////////////////////////////////////////////////////
//       if (isEvenTimestep==false)
//       {
//          D.f[DIR_P00] = &DD[DIR_P00 * size_Mat];
//          D.f[DIR_M00] = &DD[DIR_M00 * size_Mat];
//          D.f[DIR_0P0] = &DD[DIR_0P0 * size_Mat];
//          D.f[DIR_0M0] = &DD[DIR_0M0 * size_Mat];
//          D.f[DIR_00P] = &DD[DIR_00P * size_Mat];
//          D.f[DIR_00M] = &DD[DIR_00M * size_Mat];
//          D.f[DIR_PP0] = &DD[DIR_PP0 * size_Mat];
//          D.f[DIR_MM0] = &DD[DIR_MM0 * size_Mat];
//          D.f[DIR_PM0] = &DD[DIR_PM0 * size_Mat];
//          D.f[DIR_MP0] = &DD[DIR_MP0 * size_Mat];
//          D.f[DIR_P0P] = &DD[DIR_P0P * size_Mat];
//          D.f[DIR_M0M] = &DD[DIR_M0M * size_Mat];
//          D.f[DIR_P0M] = &DD[DIR_P0M * size_Mat];
//          D.f[DIR_M0P] = &DD[DIR_M0P * size_Mat];
//          D.f[DIR_0PP] = &DD[DIR_0PP * size_Mat];
//          D.f[DIR_0MM] = &DD[DIR_0MM * size_Mat];
//          D.f[DIR_0PM] = &DD[DIR_0PM * size_Mat];
//          D.f[DIR_0MP] = &DD[DIR_0MP * size_Mat];
//          D.f[DIR_000] = &DD[DIR_000 * size_Mat];
//          D.f[DIR_PPP] = &DD[DIR_PPP * size_Mat];
//          D.f[DIR_MMP] = &DD[DIR_MMP * size_Mat];
//          D.f[DIR_PMP] = &DD[DIR_PMP * size_Mat];
//          D.f[DIR_MPP] = &DD[DIR_MPP * size_Mat];
//          D.f[DIR_PPM] = &DD[DIR_PPM * size_Mat];
//          D.f[DIR_MMM] = &DD[DIR_MMM * size_Mat];
//          D.f[DIR_PMM] = &DD[DIR_PMM * size_Mat];
//          D.f[DIR_MPM] = &DD[DIR_MPM * size_Mat];
//       } 
//       else
//       {
//          D.f[DIR_M00] = &DD[DIR_P00 * size_Mat];
//          D.f[DIR_P00] = &DD[DIR_M00 * size_Mat];
//          D.f[DIR_0M0] = &DD[DIR_0P0 * size_Mat];
//          D.f[DIR_0P0] = &DD[DIR_0M0 * size_Mat];
//          D.f[DIR_00M] = &DD[DIR_00P * size_Mat];
//          D.f[DIR_00P] = &DD[DIR_00M * size_Mat];
//          D.f[DIR_MM0] = &DD[DIR_PP0 * size_Mat];
//          D.f[DIR_PP0] = &DD[DIR_MM0 * size_Mat];
//          D.f[DIR_MP0] = &DD[DIR_PM0 * size_Mat];
//          D.f[DIR_PM0] = &DD[DIR_MP0 * size_Mat];
//          D.f[DIR_M0M] = &DD[DIR_P0P * size_Mat];
//          D.f[DIR_P0P] = &DD[DIR_M0M * size_Mat];
//          D.f[DIR_M0P] = &DD[DIR_P0M * size_Mat];
//          D.f[DIR_P0M] = &DD[DIR_M0P * size_Mat];
//          D.f[DIR_0MM] = &DD[DIR_0PP * size_Mat];
//          D.f[DIR_0PP] = &DD[DIR_0MM * size_Mat];
//          D.f[DIR_0MP] = &DD[DIR_0PM * size_Mat];
//          D.f[DIR_0PM] = &DD[DIR_0MP * size_Mat];
//          D.f[DIR_000] = &DD[DIR_000 * size_Mat];
//          D.f[DIR_PPP] = &DD[DIR_MMM * size_Mat];
//          D.f[DIR_MMP] = &DD[DIR_PPM * size_Mat];
//          D.f[DIR_PMP] = &DD[DIR_MPM * size_Mat];
//          D.f[DIR_MPP] = &DD[DIR_PMM * size_Mat];
//          D.f[DIR_PPM] = &DD[DIR_MMP * size_Mat];
//          D.f[DIR_MMM] = &DD[DIR_PPP * size_Mat];
//          D.f[DIR_PMM] = &DD[DIR_MPP * size_Mat];
//          D.f[DIR_MPM] = &DD[DIR_PMP * size_Mat];
//       }
//       ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//       //Test
//       //(D.f[DIR_000])[k]=c1o10;
//       ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// 	  real om_turb = om1 / (c1o1 + c3o1*om1*max(c0o1, turbViscosity[k_Q[k]]));
     
//      real fac = c1o1;//c99o100;
// 	  real VeloX = fac*vx1;
// 	  real VeloY = fac*vx2;
// 	  real VeloZ = fac*vx3;
// 	  bool x = false;
// 	  bool y = false;
// 	  bool z = false;

//       q = q_dirE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = c0o1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 x = true;
//          feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_M00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cu_sq); 
//          //(D.f[DIR_M00])[kw]=(one-q)/(one+q)*(f_E-feq*om1)/(one-om1)+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);
//          //(D.f[DIR_M00])[kw]=zero;
//       }

//       q = q_dirW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = c0o1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 x = true;
//          feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_P00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cu_sq); 
//          //(D.f[DIR_P00])[ke]=(one-q)/(one+q)*(f_W-feq*om_turb)/(one-om_turb)+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);
//          //(D.f[DIR_P00])[ke]=zero;
//       }

//       q = q_dirN[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 		 VeloY = c0o1;
// 	     VeloZ = fac*vx3;
// 		 y = true;
//          feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cu_sq); 
//          //(D.f[DIR_0M0])[ks]=(one-q)/(one+q)*(f_N-feq*om_turb)/(one-om_turb)+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);
//          //(D.f[DIR_0M0])[ks]=zero;
//       }

//       q = q_dirS[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 		 VeloY = c0o1;
// 	     VeloZ = fac*vx3;
// 		 y = true;
//          feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cu_sq); 
//          //(D.f[DIR_0P0])[kn]=(one-q)/(one+q)*(f_S-feq*om_turb)/(one-om_turb)+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);
//          //(D.f[DIR_0P0])[kn]=zero;
//       }

//       q = q_dirT[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 		 VeloZ = c0o1;
// 		 z = true;
//          feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cu_sq); 
//          //(D.f[DIR_00M])[kb]=(one-q)/(one+q)*(f_T-feq*om_turb)/(one-om_turb)+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);
//          //(D.f[DIR_00M])[kb]=one;
//       }

//       q = q_dirB[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 		 VeloZ = c0o1;
// 		 z = true;
//          feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cu_sq); 
//          //(D.f[DIR_00P])[kt]=(one-q)/(one+q)*(f_B-feq*om_turb)/(one-om_turb)+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);
//          //(D.f[DIR_00P])[kt]=zero;
//       }

//       q = q_dirNE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
//          feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_MM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
//          //(D.f[DIR_MM0])[ksw]=(one-q)/(one+q)*(f_NE-feq*om_turb)/(one-om_turb)+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);
//          //(D.f[DIR_MM0])[ksw]=zero;
//       }

//       q = q_dirSW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
//          feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_PP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
//          //(D.f[DIR_PP0])[kne]=(one-q)/(one+q)*(f_SW-feq*om_turb)/(one-om_turb)+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);
//          //(D.f[DIR_PP0])[kne]=zero;
//       }

//       q = q_dirSE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
//          feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_MP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
//          //(D.f[DIR_MP0])[knw]=(one-q)/(one+q)*(f_SE-feq*om_turb)/(one-om_turb)+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);
//          //(D.f[DIR_MP0])[knw]=zero;
//       }

//       q = q_dirNW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
//          feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_PM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
//          //(D.f[DIR_PM0])[kse]=(one-q)/(one+q)*(f_NW-feq*om_turb)/(one-om_turb)+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);
//          //(D.f[DIR_PM0])[kse]=zero;
//       }

//       q = q_dirTE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//       //  if (k==10000) printf("AFTER x: %u \t  y: %u \t z: %u \n  VeloX: %f \t VeloY: %f \t VeloZ: %f \n\n", x,y,z, VeloX,VeloY,VeloZ);
//          feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_M0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
//          //(D.f[DIR_M0M])[kbw]=(one-q)/(one+q)*(f_TE-feq*om_turb)/(one-om_turb)+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);
//          //(D.f[DIR_M0M])[kbw]=zero;
//       }

//       q = q_dirBW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_P0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
//          //(D.f[DIR_P0P])[kte]=(one-q)/(one+q)*(f_BW-feq*om_turb)/(one-om_turb)+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);
//          //(D.f[DIR_P0P])[kte]=zero;
//       }

//       q = q_dirBE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_M0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
//          //(D.f[DIR_M0P])[ktw]=(one-q)/(one+q)*(f_BE-feq*om_turb)/(one-om_turb)+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);
//          //(D.f[DIR_M0P])[ktw]=zero;
//       }

//       q = q_dirTW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_P0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
//          //(D.f[DIR_P0M])[kbe]=(one-q)/(one+q)*(f_TW-feq*om_turb)/(one-om_turb)+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);
//          //(D.f[DIR_P0M])[kbe]=zero;
//       }

//       q = q_dirTN[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
//          //(D.f[DIR_0MM])[kbs]=(one-q)/(one+q)*(f_TN-feq*om_turb)/(one-om_turb)+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);
//          //(D.f[DIR_0MM])[kbs]=zero;
//       }

//       q = q_dirBS[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
//          //(D.f[DIR_0PP])[ktn]=(one-q)/(one+q)*(f_BS-feq*om_turb)/(one-om_turb)+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);
//          //(D.f[DIR_0PP])[ktn]=zero;
//       }

//       q = q_dirBN[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
//          //(D.f[DIR_0MP])[kts]=(one-q)/(one+q)*(f_BN-feq*om_turb)/(one-om_turb)+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);
//          //(D.f[DIR_0MP])[kts]=zero;
//       }

//       q = q_dirTS[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
//          //(D.f[DIR_0PM])[kbn]=(one-q)/(one+q)*(f_TS-feq*om_turb)/(one-om_turb)+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);
//          //(D.f[DIR_0PM])[kbn]=zero;
//       }

//       q = q_dirTNE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_MMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
//          //(D.f[DIR_MMM])[kbsw]=(one-q)/(one+q)*(f_TNE-feq*om_turb)/(one-om_turb)+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);
//          //(D.f[DIR_MMM])[kbsw]=zero;
//       }

//       q = q_dirBSW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_PPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
//          //(D.f[DIR_PPP])[ktne]=(one-q)/(one+q)*(f_BSW-feq*om_turb)/(one-om_turb)+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);
//          //(D.f[DIR_PPP])[ktne]=zero;
//       }

//       q = q_dirBNE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_MMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
//          //(D.f[DIR_MMP])[ktsw]=(one-q)/(one+q)*(f_BNE-feq*om_turb)/(one-om_turb)+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);
//          //(D.f[DIR_MMP])[ktsw]=zero;
//       }

//       q = q_dirTSW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_PPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
//          //(D.f[DIR_PPM])[kbne]=(one-q)/(one+q)*(f_TSW-feq*om_turb)/(one-om_turb)+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);
//          //(D.f[DIR_PPM])[kbne]=zero;
//       }

//       q = q_dirTSE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_MPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
//          //(D.f[DIR_MPM])[kbnw]=(one-q)/(one+q)*(f_TSE-feq*om_turb)/(one-om_turb)+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);
//          //(D.f[DIR_MPM])[kbnw]=zero;
//       }

//       q = q_dirBNW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_PMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
//          //(D.f[DIR_PMP])[ktse]=(one-q)/(one+q)*(f_BNW-feq*om_turb)/(one-om_turb)+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);
//          //(D.f[DIR_PMP])[ktse]=zero;
//       }

//       q = q_dirBSE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_MPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
//          //(D.f[DIR_MPP])[ktnw]=(one-q)/(one+q)*(f_BSE-feq*om_turb)/(one-om_turb)+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);
//          //(D.f[DIR_MPP])[ktnw]=zero;
//       }

//       q = q_dirTNW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
// 		 VeloX = fac*vx1;
// 	     VeloY = fac*vx2;
// 	     VeloZ = fac*vx3;
// 		 if (x == true) VeloX = c0o1;
// 		 if (y == true) VeloY = c0o1;
// 		 if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[DIR_PMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
//          //(D.f[DIR_PMM])[kbse]=(one-q)/(one+q)*(f_TNW-feq*om_turb)/(one-om_turb)+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);
//          //(D.f[DIR_PMM])[kbse]=zero;
//       }
//    }
// }






































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipGeomDeviceComp27(
    real* DD, 
    int* k_Q, 
    real* QQ,
    unsigned int  numberOfBCnodes,
    real om1, 
    real* NormalX,
    real* NormalY,
    real* NormalZ,
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[DIR_P00] = &DD[DIR_P00 * numberOfLBnodes];
      D.f[DIR_M00] = &DD[DIR_M00 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
   } 
   else
   {
      D.f[DIR_M00] = &DD[DIR_P00 * numberOfLBnodes];
      D.f[DIR_P00] = &DD[DIR_M00 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[DIR_P00 * numberOfBCnodes];
      q_dirW   = &QQ[DIR_M00 * numberOfBCnodes];
      q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[DIR_P00 * numberOfBCnodes];
      nx_dirW   = &NormalX[DIR_M00 * numberOfBCnodes];
      nx_dirN   = &NormalX[DIR_0P0 * numberOfBCnodes];
      nx_dirS   = &NormalX[DIR_0M0 * numberOfBCnodes];
      nx_dirT   = &NormalX[DIR_00P * numberOfBCnodes];
      nx_dirB   = &NormalX[DIR_00M * numberOfBCnodes];
      nx_dirNE  = &NormalX[DIR_PP0 * numberOfBCnodes];
      nx_dirSW  = &NormalX[DIR_MM0 * numberOfBCnodes];
      nx_dirSE  = &NormalX[DIR_PM0 * numberOfBCnodes];
      nx_dirNW  = &NormalX[DIR_MP0 * numberOfBCnodes];
      nx_dirTE  = &NormalX[DIR_P0P * numberOfBCnodes];
      nx_dirBW  = &NormalX[DIR_M0M * numberOfBCnodes];
      nx_dirBE  = &NormalX[DIR_P0M * numberOfBCnodes];
      nx_dirTW  = &NormalX[DIR_M0P * numberOfBCnodes];
      nx_dirTN  = &NormalX[DIR_0PP * numberOfBCnodes];
      nx_dirBS  = &NormalX[DIR_0MM * numberOfBCnodes];
      nx_dirBN  = &NormalX[DIR_0PM * numberOfBCnodes];
      nx_dirTS  = &NormalX[DIR_0MP * numberOfBCnodes];
      nx_dirTNE = &NormalX[DIR_PPP * numberOfBCnodes];
      nx_dirTSW = &NormalX[DIR_MMP * numberOfBCnodes];
      nx_dirTSE = &NormalX[DIR_PMP * numberOfBCnodes];
      nx_dirTNW = &NormalX[DIR_MPP * numberOfBCnodes];
      nx_dirBNE = &NormalX[DIR_PPM * numberOfBCnodes];
      nx_dirBSW = &NormalX[DIR_MMM * numberOfBCnodes];
      nx_dirBSE = &NormalX[DIR_PMM * numberOfBCnodes];
      nx_dirBNW = &NormalX[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[DIR_P00 * numberOfBCnodes];
      ny_dirW   = &NormalY[DIR_M00 * numberOfBCnodes];
      ny_dirN   = &NormalY[DIR_0P0 * numberOfBCnodes];
      ny_dirS   = &NormalY[DIR_0M0 * numberOfBCnodes];
      ny_dirT   = &NormalY[DIR_00P * numberOfBCnodes];
      ny_dirB   = &NormalY[DIR_00M * numberOfBCnodes];
      ny_dirNE  = &NormalY[DIR_PP0 * numberOfBCnodes];
      ny_dirSW  = &NormalY[DIR_MM0 * numberOfBCnodes];
      ny_dirSE  = &NormalY[DIR_PM0 * numberOfBCnodes];
      ny_dirNW  = &NormalY[DIR_MP0 * numberOfBCnodes];
      ny_dirTE  = &NormalY[DIR_P0P * numberOfBCnodes];
      ny_dirBW  = &NormalY[DIR_M0M * numberOfBCnodes];
      ny_dirBE  = &NormalY[DIR_P0M * numberOfBCnodes];
      ny_dirTW  = &NormalY[DIR_M0P * numberOfBCnodes];
      ny_dirTN  = &NormalY[DIR_0PP * numberOfBCnodes];
      ny_dirBS  = &NormalY[DIR_0MM * numberOfBCnodes];
      ny_dirBN  = &NormalY[DIR_0PM * numberOfBCnodes];
      ny_dirTS  = &NormalY[DIR_0MP * numberOfBCnodes];
      ny_dirTNE = &NormalY[DIR_PPP * numberOfBCnodes];
      ny_dirTSW = &NormalY[DIR_MMP * numberOfBCnodes];
      ny_dirTSE = &NormalY[DIR_PMP * numberOfBCnodes];
      ny_dirTNW = &NormalY[DIR_MPP * numberOfBCnodes];
      ny_dirBNE = &NormalY[DIR_PPM * numberOfBCnodes];
      ny_dirBSW = &NormalY[DIR_MMM * numberOfBCnodes];
      ny_dirBSE = &NormalY[DIR_PMM * numberOfBCnodes];
      ny_dirBNW = &NormalY[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[DIR_P00 * numberOfBCnodes];
      nz_dirW   = &NormalZ[DIR_M00 * numberOfBCnodes];
      nz_dirN   = &NormalZ[DIR_0P0 * numberOfBCnodes];
      nz_dirS   = &NormalZ[DIR_0M0 * numberOfBCnodes];
      nz_dirT   = &NormalZ[DIR_00P * numberOfBCnodes];
      nz_dirB   = &NormalZ[DIR_00M * numberOfBCnodes];
      nz_dirNE  = &NormalZ[DIR_PP0 * numberOfBCnodes];
      nz_dirSW  = &NormalZ[DIR_MM0 * numberOfBCnodes];
      nz_dirSE  = &NormalZ[DIR_PM0 * numberOfBCnodes];
      nz_dirNW  = &NormalZ[DIR_MP0 * numberOfBCnodes];
      nz_dirTE  = &NormalZ[DIR_P0P * numberOfBCnodes];
      nz_dirBW  = &NormalZ[DIR_M0M * numberOfBCnodes];
      nz_dirBE  = &NormalZ[DIR_P0M * numberOfBCnodes];
      nz_dirTW  = &NormalZ[DIR_M0P * numberOfBCnodes];
      nz_dirTN  = &NormalZ[DIR_0PP * numberOfBCnodes];
      nz_dirBS  = &NormalZ[DIR_0MM * numberOfBCnodes];
      nz_dirBN  = &NormalZ[DIR_0PM * numberOfBCnodes];
      nz_dirTS  = &NormalZ[DIR_0MP * numberOfBCnodes];
      nz_dirTNE = &NormalZ[DIR_PPP * numberOfBCnodes];
      nz_dirTSW = &NormalZ[DIR_MMP * numberOfBCnodes];
      nz_dirTSE = &NormalZ[DIR_PMP * numberOfBCnodes];
      nz_dirTNW = &NormalZ[DIR_MPP * numberOfBCnodes];
      nz_dirBNE = &NormalZ[DIR_PPM * numberOfBCnodes];
      nz_dirBSW = &NormalZ[DIR_MMM * numberOfBCnodes];
      nz_dirBSE = &NormalZ[DIR_PMM * numberOfBCnodes];
      nz_dirBNW = &NormalZ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[DIR_P00])[ke   ];
      real f_E    = (D.f[DIR_M00])[kw   ];
      real f_S    = (D.f[DIR_0P0])[kn   ];
      real f_N    = (D.f[DIR_0M0])[ks   ];
      real f_B    = (D.f[DIR_00P])[kt   ];
      real f_T    = (D.f[DIR_00M])[kb   ];
      real f_SW   = (D.f[DIR_PP0])[kne  ];
      real f_NE   = (D.f[DIR_MM0])[ksw  ];
      real f_NW   = (D.f[DIR_PM0])[kse  ];
      real f_SE   = (D.f[DIR_MP0])[knw  ];
      real f_BW   = (D.f[DIR_P0P])[kte  ];
      real f_TE   = (D.f[DIR_M0M])[kbw  ];
      real f_TW   = (D.f[DIR_P0M])[kbe  ];
      real f_BE   = (D.f[DIR_M0P])[ktw  ];
      real f_BS   = (D.f[DIR_0PP])[ktn  ];
      real f_TN   = (D.f[DIR_0MM])[kbs  ];
      real f_TS   = (D.f[DIR_0PM])[kbn  ];
      real f_BN   = (D.f[DIR_0MP])[kts  ];
      real f_BSW  = (D.f[DIR_PPP])[ktne ];
      real f_BNE  = (D.f[DIR_MMP])[ktsw ];
      real f_BNW  = (D.f[DIR_PMP])[ktse ];
      real f_BSE  = (D.f[DIR_MPP])[ktnw ];
      real f_TSW  = (D.f[DIR_PPM])[kbne ];
      real f_TNE  = (D.f[DIR_MMM])[kbsw ];
      real f_TNW  = (D.f[DIR_PMM])[kbse ];
      real f_TSE  = (D.f[DIR_MPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[DIR_000])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[DIR_P00] = &DD[DIR_P00 * numberOfLBnodes];
         D.f[DIR_M00] = &DD[DIR_M00 * numberOfLBnodes];
         D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
         D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
         D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
         D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
         D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
         D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
         D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
         D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
         D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
         D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
         D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
         D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
         D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
         D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
         D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
         D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
         D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
         D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
         D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
         D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
         D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
         D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
         D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
         D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
      } 
      else
      {
         D.f[DIR_M00] = &DD[DIR_P00 * numberOfLBnodes];
         D.f[DIR_P00] = &DD[DIR_M00 * numberOfLBnodes];
         D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
         D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
         D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
         D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
         D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
         D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
         D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
         D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
         D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
         D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
         D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
         D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
         D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
         D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
         D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
         D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
         D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
         D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
         D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
         D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
         D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
         D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
         D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
         D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real VeloX = vx1;
	  real VeloY = vx2;
	  real VeloZ = vx3;
	  real fac = c0o1;//0.5;
 	  real phi = c0o1;
	  //real alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real kxyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho) - ((vx1*vx2)));
      real kyzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho) - ((vx2*vx3)));
      real kxzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho) - ((vx1*vx3)));

	  real kxxFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx1*vx1)));
	  real kyyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx2*vx2)));
	  real kzzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx3*vx3)));

	  real magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

	  //fac = fac * magS / (c1o3 * (one / om1 - c1o2));
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real *facAst = &QQ[DIR_000 * numberOfBCnodes];

	  //fac = fac * alpha + facAst[k] * (one - alpha);
	  //facAst[k] = fac;
	  //(&QQ[DIR_000 * numberOfBCnodes])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////real uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real phi = expf(magS/0.01f) - one;
	  //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real C = five;
	  //real kappa = 0.41f;
	  //real phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
	  //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
	  //real qMed = sum/count;
	  //real phi = fac / (qMed + fac);
	  //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real testQ = c2o1;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirE[k]) + fac);
		 VeloX *= phi;
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_M00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirW[k]) + fac);
		 VeloX *= phi;
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_P00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirN[k]) + fac);
		 VeloY *= phi;
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirS[k]) + fac);
		 VeloY *= phi;
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nz_dirT[k]) + fac);
		 VeloZ *= phi;
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nz_dirB[k]) + fac);
		 VeloZ *= phi;
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
		 VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirNE[k] + ny_dirNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
		 VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirSW[k] - ny_dirSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
		 VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirSE[k] - ny_dirSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
		 VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirNW[k] + ny_dirNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTE[k] + nz_dirTE[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_M0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBW[k] - nz_dirBW[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_P0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBE[k] - nz_dirBE[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_M0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTW[k] + nz_dirTW[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_P0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
		 VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirTN[k] + nz_dirTN[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
		 VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirBS[k] - nz_dirBS[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
		 VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirBN[k] - nz_dirBN[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
		 VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirTS[k] + nz_dirTS[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
		 VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
		 VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
		 VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
		 VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
		 VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
		 VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
		 VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = (phi > one) ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
		 VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipNormDeviceComp27(
    real* DD, 
    int* k_Q, 
    real* QQ,
    unsigned int  numberOfBCnodes,
    real om1, 
    real* NormalX,
    real* NormalY,
    real* NormalZ,
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[DIR_P00] = &DD[DIR_P00 * numberOfLBnodes];
      D.f[DIR_M00] = &DD[DIR_M00 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
   } 
   else
   {
      D.f[DIR_M00] = &DD[DIR_P00 * numberOfLBnodes];
      D.f[DIR_P00] = &DD[DIR_M00 * numberOfLBnodes];
      D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
      D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
      D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
      D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
      D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
      D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
      D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
      D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
      D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
      D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
      D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
      D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
      D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
      D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
      D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
      D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
      D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
      D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
      D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
      D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
      D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
      D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
      D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
      D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
      D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[DIR_P00 * numberOfBCnodes];
      q_dirW   = &QQ[DIR_M00 * numberOfBCnodes];
      q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[DIR_P00 * numberOfBCnodes];
      nx_dirW   = &NormalX[DIR_M00 * numberOfBCnodes];
      nx_dirN   = &NormalX[DIR_0P0 * numberOfBCnodes];
      nx_dirS   = &NormalX[DIR_0M0 * numberOfBCnodes];
      nx_dirT   = &NormalX[DIR_00P * numberOfBCnodes];
      nx_dirB   = &NormalX[DIR_00M * numberOfBCnodes];
      nx_dirNE  = &NormalX[DIR_PP0 * numberOfBCnodes];
      nx_dirSW  = &NormalX[DIR_MM0 * numberOfBCnodes];
      nx_dirSE  = &NormalX[DIR_PM0 * numberOfBCnodes];
      nx_dirNW  = &NormalX[DIR_MP0 * numberOfBCnodes];
      nx_dirTE  = &NormalX[DIR_P0P * numberOfBCnodes];
      nx_dirBW  = &NormalX[DIR_M0M * numberOfBCnodes];
      nx_dirBE  = &NormalX[DIR_P0M * numberOfBCnodes];
      nx_dirTW  = &NormalX[DIR_M0P * numberOfBCnodes];
      nx_dirTN  = &NormalX[DIR_0PP * numberOfBCnodes];
      nx_dirBS  = &NormalX[DIR_0MM * numberOfBCnodes];
      nx_dirBN  = &NormalX[DIR_0PM * numberOfBCnodes];
      nx_dirTS  = &NormalX[DIR_0MP * numberOfBCnodes];
      nx_dirTNE = &NormalX[DIR_PPP * numberOfBCnodes];
      nx_dirTSW = &NormalX[DIR_MMP * numberOfBCnodes];
      nx_dirTSE = &NormalX[DIR_PMP * numberOfBCnodes];
      nx_dirTNW = &NormalX[DIR_MPP * numberOfBCnodes];
      nx_dirBNE = &NormalX[DIR_PPM * numberOfBCnodes];
      nx_dirBSW = &NormalX[DIR_MMM * numberOfBCnodes];
      nx_dirBSE = &NormalX[DIR_PMM * numberOfBCnodes];
      nx_dirBNW = &NormalX[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[DIR_P00 * numberOfBCnodes];
      ny_dirW   = &NormalY[DIR_M00 * numberOfBCnodes];
      ny_dirN   = &NormalY[DIR_0P0 * numberOfBCnodes];
      ny_dirS   = &NormalY[DIR_0M0 * numberOfBCnodes];
      ny_dirT   = &NormalY[DIR_00P * numberOfBCnodes];
      ny_dirB   = &NormalY[DIR_00M * numberOfBCnodes];
      ny_dirNE  = &NormalY[DIR_PP0 * numberOfBCnodes];
      ny_dirSW  = &NormalY[DIR_MM0 * numberOfBCnodes];
      ny_dirSE  = &NormalY[DIR_PM0 * numberOfBCnodes];
      ny_dirNW  = &NormalY[DIR_MP0 * numberOfBCnodes];
      ny_dirTE  = &NormalY[DIR_P0P * numberOfBCnodes];
      ny_dirBW  = &NormalY[DIR_M0M * numberOfBCnodes];
      ny_dirBE  = &NormalY[DIR_P0M * numberOfBCnodes];
      ny_dirTW  = &NormalY[DIR_M0P * numberOfBCnodes];
      ny_dirTN  = &NormalY[DIR_0PP * numberOfBCnodes];
      ny_dirBS  = &NormalY[DIR_0MM * numberOfBCnodes];
      ny_dirBN  = &NormalY[DIR_0PM * numberOfBCnodes];
      ny_dirTS  = &NormalY[DIR_0MP * numberOfBCnodes];
      ny_dirTNE = &NormalY[DIR_PPP * numberOfBCnodes];
      ny_dirTSW = &NormalY[DIR_MMP * numberOfBCnodes];
      ny_dirTSE = &NormalY[DIR_PMP * numberOfBCnodes];
      ny_dirTNW = &NormalY[DIR_MPP * numberOfBCnodes];
      ny_dirBNE = &NormalY[DIR_PPM * numberOfBCnodes];
      ny_dirBSW = &NormalY[DIR_MMM * numberOfBCnodes];
      ny_dirBSE = &NormalY[DIR_PMM * numberOfBCnodes];
      ny_dirBNW = &NormalY[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[DIR_P00 * numberOfBCnodes];
      nz_dirW   = &NormalZ[DIR_M00 * numberOfBCnodes];
      nz_dirN   = &NormalZ[DIR_0P0 * numberOfBCnodes];
      nz_dirS   = &NormalZ[DIR_0M0 * numberOfBCnodes];
      nz_dirT   = &NormalZ[DIR_00P * numberOfBCnodes];
      nz_dirB   = &NormalZ[DIR_00M * numberOfBCnodes];
      nz_dirNE  = &NormalZ[DIR_PP0 * numberOfBCnodes];
      nz_dirSW  = &NormalZ[DIR_MM0 * numberOfBCnodes];
      nz_dirSE  = &NormalZ[DIR_PM0 * numberOfBCnodes];
      nz_dirNW  = &NormalZ[DIR_MP0 * numberOfBCnodes];
      nz_dirTE  = &NormalZ[DIR_P0P * numberOfBCnodes];
      nz_dirBW  = &NormalZ[DIR_M0M * numberOfBCnodes];
      nz_dirBE  = &NormalZ[DIR_P0M * numberOfBCnodes];
      nz_dirTW  = &NormalZ[DIR_M0P * numberOfBCnodes];
      nz_dirTN  = &NormalZ[DIR_0PP * numberOfBCnodes];
      nz_dirBS  = &NormalZ[DIR_0MM * numberOfBCnodes];
      nz_dirBN  = &NormalZ[DIR_0PM * numberOfBCnodes];
      nz_dirTS  = &NormalZ[DIR_0MP * numberOfBCnodes];
      nz_dirTNE = &NormalZ[DIR_PPP * numberOfBCnodes];
      nz_dirTSW = &NormalZ[DIR_MMP * numberOfBCnodes];
      nz_dirTSE = &NormalZ[DIR_PMP * numberOfBCnodes];
      nz_dirTNW = &NormalZ[DIR_MPP * numberOfBCnodes];
      nz_dirBNE = &NormalZ[DIR_PPM * numberOfBCnodes];
      nz_dirBSW = &NormalZ[DIR_MMM * numberOfBCnodes];
      nz_dirBSE = &NormalZ[DIR_PMM * numberOfBCnodes];
      nz_dirBNW = &NormalZ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[DIR_P00])[ke   ];
      real f_E    = (D.f[DIR_M00])[kw   ];
      real f_S    = (D.f[DIR_0P0])[kn   ];
      real f_N    = (D.f[DIR_0M0])[ks   ];
      real f_B    = (D.f[DIR_00P])[kt   ];
      real f_T    = (D.f[DIR_00M])[kb   ];
      real f_SW   = (D.f[DIR_PP0])[kne  ];
      real f_NE   = (D.f[DIR_MM0])[ksw  ];
      real f_NW   = (D.f[DIR_PM0])[kse  ];
      real f_SE   = (D.f[DIR_MP0])[knw  ];
      real f_BW   = (D.f[DIR_P0P])[kte  ];
      real f_TE   = (D.f[DIR_M0M])[kbw  ];
      real f_TW   = (D.f[DIR_P0M])[kbe  ];
      real f_BE   = (D.f[DIR_M0P])[ktw  ];
      real f_BS   = (D.f[DIR_0PP])[ktn  ];
      real f_TN   = (D.f[DIR_0MM])[kbs  ];
      real f_TS   = (D.f[DIR_0PM])[kbn  ];
      real f_BN   = (D.f[DIR_0MP])[kts  ];
      real f_BSW  = (D.f[DIR_PPP])[ktne ];
      real f_BNE  = (D.f[DIR_MMP])[ktsw ];
      real f_BNW  = (D.f[DIR_PMP])[ktse ];
      real f_BSE  = (D.f[DIR_MPP])[ktnw ];
      real f_TSW  = (D.f[DIR_PPM])[kbne ];
      real f_TNE  = (D.f[DIR_MMM])[kbsw ];
      real f_TNW  = (D.f[DIR_PMM])[kbse ];
      real f_TSE  = (D.f[DIR_MPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[DIR_000])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[DIR_P00] = &DD[DIR_P00 * numberOfLBnodes];
         D.f[DIR_M00] = &DD[DIR_M00 * numberOfLBnodes];
         D.f[DIR_0P0] = &DD[DIR_0P0 * numberOfLBnodes];
         D.f[DIR_0M0] = &DD[DIR_0M0 * numberOfLBnodes];
         D.f[DIR_00P] = &DD[DIR_00P * numberOfLBnodes];
         D.f[DIR_00M] = &DD[DIR_00M * numberOfLBnodes];
         D.f[DIR_PP0] = &DD[DIR_PP0 * numberOfLBnodes];
         D.f[DIR_MM0] = &DD[DIR_MM0 * numberOfLBnodes];
         D.f[DIR_PM0] = &DD[DIR_PM0 * numberOfLBnodes];
         D.f[DIR_MP0] = &DD[DIR_MP0 * numberOfLBnodes];
         D.f[DIR_P0P] = &DD[DIR_P0P * numberOfLBnodes];
         D.f[DIR_M0M] = &DD[DIR_M0M * numberOfLBnodes];
         D.f[DIR_P0M] = &DD[DIR_P0M * numberOfLBnodes];
         D.f[DIR_M0P] = &DD[DIR_M0P * numberOfLBnodes];
         D.f[DIR_0PP] = &DD[DIR_0PP * numberOfLBnodes];
         D.f[DIR_0MM] = &DD[DIR_0MM * numberOfLBnodes];
         D.f[DIR_0PM] = &DD[DIR_0PM * numberOfLBnodes];
         D.f[DIR_0MP] = &DD[DIR_0MP * numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
         D.f[DIR_PPP] = &DD[DIR_PPP * numberOfLBnodes];
         D.f[DIR_MMP] = &DD[DIR_MMP * numberOfLBnodes];
         D.f[DIR_PMP] = &DD[DIR_PMP * numberOfLBnodes];
         D.f[DIR_MPP] = &DD[DIR_MPP * numberOfLBnodes];
         D.f[DIR_PPM] = &DD[DIR_PPM * numberOfLBnodes];
         D.f[DIR_MMM] = &DD[DIR_MMM * numberOfLBnodes];
         D.f[DIR_PMM] = &DD[DIR_PMM * numberOfLBnodes];
         D.f[DIR_MPM] = &DD[DIR_MPM * numberOfLBnodes];
      } 
      else
      {
         D.f[DIR_M00] = &DD[DIR_P00 * numberOfLBnodes];
         D.f[DIR_P00] = &DD[DIR_M00 * numberOfLBnodes];
         D.f[DIR_0M0] = &DD[DIR_0P0 * numberOfLBnodes];
         D.f[DIR_0P0] = &DD[DIR_0M0 * numberOfLBnodes];
         D.f[DIR_00M] = &DD[DIR_00P * numberOfLBnodes];
         D.f[DIR_00P] = &DD[DIR_00M * numberOfLBnodes];
         D.f[DIR_MM0] = &DD[DIR_PP0 * numberOfLBnodes];
         D.f[DIR_PP0] = &DD[DIR_MM0 * numberOfLBnodes];
         D.f[DIR_MP0] = &DD[DIR_PM0 * numberOfLBnodes];
         D.f[DIR_PM0] = &DD[DIR_MP0 * numberOfLBnodes];
         D.f[DIR_M0M] = &DD[DIR_P0P * numberOfLBnodes];
         D.f[DIR_P0P] = &DD[DIR_M0M * numberOfLBnodes];
         D.f[DIR_M0P] = &DD[DIR_P0M * numberOfLBnodes];
         D.f[DIR_P0M] = &DD[DIR_M0P * numberOfLBnodes];
         D.f[DIR_0MM] = &DD[DIR_0PP * numberOfLBnodes];
         D.f[DIR_0PP] = &DD[DIR_0MM * numberOfLBnodes];
         D.f[DIR_0MP] = &DD[DIR_0PM * numberOfLBnodes];
         D.f[DIR_0PM] = &DD[DIR_0MP * numberOfLBnodes];
         D.f[DIR_000] = &DD[DIR_000 * numberOfLBnodes];
         D.f[DIR_PPP] = &DD[DIR_MMM * numberOfLBnodes];
         D.f[DIR_MMP] = &DD[DIR_PPM * numberOfLBnodes];
         D.f[DIR_PMP] = &DD[DIR_MPM * numberOfLBnodes];
         D.f[DIR_MPP] = &DD[DIR_PMM * numberOfLBnodes];
         D.f[DIR_PPM] = &DD[DIR_MMP * numberOfLBnodes];
         D.f[DIR_MMM] = &DD[DIR_PPP * numberOfLBnodes];
         D.f[DIR_PMM] = &DD[DIR_MPP * numberOfLBnodes];
         D.f[DIR_MPM] = &DD[DIR_PMP * numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real VeloX = vx1;
	  real VeloY = vx2;
	  real VeloZ = vx3;
	  real fac = c1o100;//0.5;
 	  //real phi = c0o1;
	  real alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real kxyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho) - ((vx1*vx2)));
      real kyzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho) - ((vx2*vx3)));
      real kxzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho) - ((vx1*vx3)));

	  real kxxFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx1*vx1)));
	  real kyyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx2*vx2)));
	  real kzzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx3*vx3)));

	  real magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

	  fac = fac * magS / (c1o3 * (c1o1 / om1 - c1o2));
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real *facAst = &QQ[DIR_000 * numberOfBCnodes];

	  fac = fac * alpha + facAst[k] * (c1o1 - alpha);
	  facAst[k] = fac;
	  //(&QQ[DIR_000 * numberOfBCnodes])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////real uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real phi = expf(magS/0.01f) - one;
	  //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real C = five;
	  //real kappa = 0.41f;
	  //real phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
	  //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
	  //real qMed = sum/count;
	  //real phi = fac / (qMed + fac);
	  //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real sliplength = 0.9f;//c1o2;
	  real qSlip = c0o1;
	  real un = c0o1;
	  real ut = c0o1;
	  real tangential = c0o1;
	  //real smallSingle = Op0000002;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
		 un = fabs((vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k]);
		 ut = fabs(VeloX);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_M00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
		 un = fabs(-(vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k]);
		 ut = fabs(-VeloX);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_P00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
		 un = fabs( (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k]);
		 ut = fabs( VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
		 un = fabs(-(vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k]);
		 ut = fabs(-VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
		 un = fabs( (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k]);
		 ut = fabs( VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nz_dirT[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
		 un = fabs(-(vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k]);
		 ut = fabs(-VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nz_dirB[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
		 VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
		 un = fabs( (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k] + (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k]);
		 ut = fabs( VeloX + VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirNE[k]+ny_dirNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
		 VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
		 un = fabs(-(vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k] - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k]);
		 ut = fabs(-VeloX - VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirSW[k]-ny_dirSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
		 VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
		 un = fabs( (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k] - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k]);
		 ut = fabs( VeloX - VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirSE[k]-ny_dirSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
		 VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
		 un = fabs(-(vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k] + (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k]);
		 ut = fabs(-VeloX + VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirNW[k]+ny_dirNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
		 un = fabs( (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k] + (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k]);
		 ut = fabs( VeloX + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTE[k]+nz_dirTE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_M0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
		 un = fabs(-(vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k] - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k]);
		 ut = fabs(-VeloX - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBW[k]-nz_dirBW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_P0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
		 un = fabs( (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k] - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k]);
		 ut = fabs( VeloX - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBE[k]-nz_dirBE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_M0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
		 un = fabs(-(vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k] + (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k]);
		 ut = fabs(-VeloX + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTW[k]+nz_dirTW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_P0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
		 VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
		 un = fabs( (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k] + (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k]);
		 ut = fabs( VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirTN[k]+nz_dirTN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
		 VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
		 un = fabs(-(vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k] - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k]);
		 ut = fabs(-VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirBS[k]-nz_dirBS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
		 VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
		 un = fabs( (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k] - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k]);
		 ut = fabs( VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirBN[k]-nz_dirBN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
		 VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
		 un = fabs(-(vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k] + (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k]);
		 ut = fabs(-VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirTS[k]+nz_dirTS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
		 VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
		 un = fabs( (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k] 
				   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k] 
				   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k]);
		 ut = fabs( VeloX + VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
		 VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
		 un = fabs(-(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k] 
				   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k] 
				   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k]);
		 ut = fabs(-VeloX - VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
		 VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
		 un = fabs( (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k] 
				   +(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k] 
				   -(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k]);
		 ut = fabs( VeloX + VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
		 VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
		 un = fabs(-(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k] 
				   -(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k] 
				   +(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k]);
		 ut = fabs(-VeloX - VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
		 VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
		 un = fabs(+(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k] 
				   -(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k] 
				   +(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k]);
		 ut = fabs(+VeloX - VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
		 VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
		 un = fabs(-(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k] 
				   +(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k] 
				   -(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k]);
		 ut = fabs(-VeloX + VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
		 VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
		 un = fabs( (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k] 
				   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k] 
				   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k]);
		 ut = fabs( VeloX - VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_MPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
		 VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
		 un = fabs(-(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k] 
				   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k] 
				   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k]);
		 ut = fabs(-VeloX + VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[DIR_PMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q) - c1o216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
