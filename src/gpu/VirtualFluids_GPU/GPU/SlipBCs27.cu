#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;

//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipDevice27(real* DD, 
                                         int* k_Q, 
                                         real* QQ,
                                         unsigned int sizeQ,
                                         real om1, 
                                         unsigned int* neighborX,
                                         unsigned int* neighborY,
                                         unsigned int* neighborZ,
                                         unsigned int size_Mat, 
                                         bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[dirE   ])[ke   ];
      real f_E    = (D.f[dirW   ])[kw   ];
      real f_S    = (D.f[dirN   ])[kn   ];
      real f_N    = (D.f[dirS   ])[ks   ];
      real f_B    = (D.f[dirT   ])[kt   ];
      real f_T    = (D.f[dirB   ])[kb   ];
      real f_SW   = (D.f[dirNE  ])[kne  ];
      real f_NE   = (D.f[dirSW  ])[ksw  ];
      real f_NW   = (D.f[dirSE  ])[kse  ];
      real f_SE   = (D.f[dirNW  ])[knw  ];
      real f_BW   = (D.f[dirTE  ])[kte  ];
      real f_TE   = (D.f[dirBW  ])[kbw  ];
      real f_TW   = (D.f[dirBE  ])[kbe  ];
      real f_BE   = (D.f[dirTW  ])[ktw  ];
      real f_BS   = (D.f[dirTN  ])[ktn  ];
      real f_TN   = (D.f[dirBS  ])[kbs  ];
      real f_TS   = (D.f[dirBN  ])[kbn  ];
      real f_BN   = (D.f[dirTS  ])[kts  ];
      real f_BSW  = (D.f[dirTNE ])[ktne ];
      real f_BNE  = (D.f[dirTSW ])[ktsw ];
      real f_BNW  = (D.f[dirTSE ])[ktse ];
      real f_BSE  = (D.f[dirTNW ])[ktnw ];
      real f_TSW  = (D.f[dirBNE ])[kbne ];
      real f_TNE  = (D.f[dirBSW ])[kbsw ];
      real f_TNW  = (D.f[dirBSE ])[kbse ];
      real f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  ((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W); 
         

      vx2    =   (-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S); 

      vx3    =   ((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real fac = c1o1;//c99o100;
	  real VeloX = fac*vx1;
	  real VeloY = fac*vx2;
	  real VeloZ = fac*vx3;
	  bool x = false;
	  bool y = false;
	  bool z = false;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho+c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-feq*om1)/(c1o1-om1)+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho+c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-feq*om1)/(c1o1-om1)+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho+c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-feq*om1)/(c1o1-om1)+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho+c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-feq*om1)/(c1o1-om1)+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho+c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-feq*om1)/(c1o1-om1)+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho+c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-feq*om1)/(c1o1-om1)+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-feq*om1)/(c1o1-om1)+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-feq*om1)/(c1o1-om1)+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-feq*om1)/(c1o1-om1)+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-feq*om1)/(c1o1-om1)+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-feq*om1)/(c1o1-om1)+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-feq*om1)/(c1o1-om1)+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-feq*om1)/(c1o1-om1)+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-feq*om1)/(c1o1-om1)+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-feq*om1)/(c1o1-om1)+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-feq*om1)/(c1o1-om1)+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-feq*om1)/(c1o1-om1)+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-feq*om1)/(c1o1-om1)+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-feq*om1)/(c1o1-om1)+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-feq*om1)/(c1o1-om1)+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-feq*om1)/(c1o1-om1)+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-feq*om1)/(c1o1-om1)+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-feq*om1)/(c1o1-om1)+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-feq*om1)/(c1o1-om1)+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-feq*om1)/(c1o1-om1)+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-feq*om1)/(c1o1-om1)+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipDeviceComp27(real* DD, 
											 int* k_Q, 
											 real* QQ,
											 unsigned int sizeQ,
											 real om1, 
											 unsigned int* neighborX,
											 unsigned int* neighborY,
											 unsigned int* neighborZ,
											 unsigned int size_Mat, 
											 bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[dirE   ])[ke   ];
      real f_E    = (D.f[dirW   ])[kw   ];
      real f_S    = (D.f[dirN   ])[kn   ];
      real f_N    = (D.f[dirS   ])[ks   ];
      real f_B    = (D.f[dirT   ])[kt   ];
      real f_T    = (D.f[dirB   ])[kb   ];
      real f_SW   = (D.f[dirNE  ])[kne  ];
      real f_NE   = (D.f[dirSW  ])[ksw  ];
      real f_NW   = (D.f[dirSE  ])[kse  ];
      real f_SE   = (D.f[dirNW  ])[knw  ];
      real f_BW   = (D.f[dirTE  ])[kte  ];
      real f_TE   = (D.f[dirBW  ])[kbw  ];
      real f_TW   = (D.f[dirBE  ])[kbe  ];
      real f_BE   = (D.f[dirTW  ])[ktw  ];
      real f_BS   = (D.f[dirTN  ])[ktn  ];
      real f_TN   = (D.f[dirBS  ])[kbs  ];
      real f_TS   = (D.f[dirBN  ])[kbn  ];
      real f_BN   = (D.f[dirTS  ])[kts  ];
      real f_BSW  = (D.f[dirTNE ])[ktne ];
      real f_BNE  = (D.f[dirTSW ])[ktsw ];
      real f_BNW  = (D.f[dirTSE ])[ktse ];
      real f_BSE  = (D.f[dirTNW ])[ktnw ];
      real f_TSW  = (D.f[dirBNE ])[kbne ];
      real f_TNE  = (D.f[dirBSW ])[kbsw ];
      real f_TNW  = (D.f[dirBSE ])[kbse ];
      real f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real fac = c1o1;//c99o100;
	  real VeloX = fac*vx1;
	  real VeloY = fac*vx2;
	  real VeloZ = fac*vx3;
	  bool x = false;
	  bool y = false;
	  bool z = false;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cu_sq); 
         //(D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-feq*om1)/(one-om1)+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cu_sq); 
         //(D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-feq*om1)/(one-om1)+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cu_sq); 
         //(D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-feq*om1)/(one-om1)+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cu_sq); 
         //(D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-feq*om1)/(one-om1)+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cu_sq); 
         //(D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-feq*om1)/(one-om1)+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cu_sq); 
         //(D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-feq*om1)/(one-om1)+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         //(D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-feq*om1)/(one-om1)+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         //(D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-feq*om1)/(one-om1)+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         //(D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-feq*om1)/(one-om1)+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         //(D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-feq*om1)/(one-om1)+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
      //  if (k==10000) printf("AFTER x: %u \t  y: %u \t z: %u \n  VeloX: %f \t VeloY: %f \t VeloZ: %f \n\n", x,y,z, VeloX,VeloY,VeloZ);
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         //(D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-feq*om1)/(one-om1)+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         //(D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-feq*om1)/(one-om1)+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         //(D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-feq*om1)/(one-om1)+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         //(D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-feq*om1)/(one-om1)+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         //(D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-feq*om1)/(one-om1)+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         //(D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-feq*om1)/(one-om1)+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         //(D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-feq*om1)/(one-om1)+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         //(D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-feq*om1)/(one-om1)+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         //(D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-feq*om1)/(one-om1)+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         //(D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-feq*om1)/(one-om1)+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         //(D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-feq*om1)/(one-om1)+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         //(D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-feq*om1)/(one-om1)+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         //(D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-feq*om1)/(one-om1)+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         //(D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-feq*om1)/(one-om1)+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         //(D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-feq*om1)/(one-om1)+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         //(D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-feq*om1)/(one-om1)+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipDeviceComp27TurbViscosity(real* DD, 
											 int* k_Q, 
											 real* QQ,
											 unsigned int sizeQ,
											 real om1, 
											 unsigned int* neighborX,
											 unsigned int* neighborY,
											 unsigned int* neighborZ,
                                  real* turbViscosity,
											 unsigned int size_Mat, 
											 bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[dirE   ])[ke   ];
      real f_E    = (D.f[dirW   ])[kw   ];
      real f_S    = (D.f[dirN   ])[kn   ];
      real f_N    = (D.f[dirS   ])[ks   ];
      real f_B    = (D.f[dirT   ])[kt   ];
      real f_T    = (D.f[dirB   ])[kb   ];
      real f_SW   = (D.f[dirNE  ])[kne  ];
      real f_NE   = (D.f[dirSW  ])[ksw  ];
      real f_NW   = (D.f[dirSE  ])[kse  ];
      real f_SE   = (D.f[dirNW  ])[knw  ];
      real f_BW   = (D.f[dirTE  ])[kte  ];
      real f_TE   = (D.f[dirBW  ])[kbw  ];
      real f_TW   = (D.f[dirBE  ])[kbe  ];
      real f_BE   = (D.f[dirTW  ])[ktw  ];
      real f_BS   = (D.f[dirTN  ])[ktn  ];
      real f_TN   = (D.f[dirBS  ])[kbs  ];
      real f_TS   = (D.f[dirBN  ])[kbn  ];
      real f_BN   = (D.f[dirTS  ])[kts  ];
      real f_BSW  = (D.f[dirTNE ])[ktne ];
      real f_BNE  = (D.f[dirTSW ])[ktsw ];
      real f_BNW  = (D.f[dirTSE ])[ktse ];
      real f_BSE  = (D.f[dirTNW ])[ktnw ];
      real f_TSW  = (D.f[dirBNE ])[kbne ];
      real f_TNE  = (D.f[dirBSW ])[kbsw ];
      real f_TNW  = (D.f[dirBSE ])[kbse ];
      real f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real om_turb = om1 / (c1o1 + c3o1*om1*max(c0o1, turbViscosity[k_Q[k]]));
     
     real fac = c1o1;//c99o100;
	  real VeloX = fac*vx1;
	  real VeloY = fac*vx2;
	  real VeloZ = fac*vx3;
	  bool x = false;
	  bool y = false;
	  bool z = false;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cu_sq); 
         //(D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-feq*om1)/(one-om1)+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cu_sq); 
         //(D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-feq*om_turb)/(one-om_turb)+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cu_sq); 
         //(D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-feq*om_turb)/(one-om_turb)+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cu_sq); 
         //(D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-feq*om_turb)/(one-om_turb)+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cu_sq); 
         //(D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-feq*om_turb)/(one-om_turb)+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cu_sq); 
         //(D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-feq*om_turb)/(one-om_turb)+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         //(D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-feq*om_turb)/(one-om_turb)+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         //(D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-feq*om_turb)/(one-om_turb)+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         //(D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-feq*om_turb)/(one-om_turb)+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         //(D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-feq*om_turb)/(one-om_turb)+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
      //  if (k==10000) printf("AFTER x: %u \t  y: %u \t z: %u \n  VeloX: %f \t VeloY: %f \t VeloZ: %f \n\n", x,y,z, VeloX,VeloY,VeloZ);
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         //(D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-feq*om_turb)/(one-om_turb)+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         //(D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-feq*om_turb)/(one-om_turb)+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         //(D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-feq*om_turb)/(one-om_turb)+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         //(D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-feq*om_turb)/(one-om_turb)+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         //(D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-feq*om_turb)/(one-om_turb)+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         //(D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-feq*om_turb)/(one-om_turb)+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         //(D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-feq*om_turb)/(one-om_turb)+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         //(D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-feq*om_turb)/(one-om_turb)+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         //(D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-feq*om_turb)/(one-om_turb)+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         //(D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-feq*om_turb)/(one-om_turb)+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         //(D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-feq*om_turb)/(one-om_turb)+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         //(D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-feq*om_turb)/(one-om_turb)+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         //(D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-feq*om_turb)/(one-om_turb)+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         //(D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-feq*om_turb)/(one-om_turb)+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         //(D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-feq*om_turb)/(one-om_turb)+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         //(D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-feq*om_turb)/(one-om_turb)+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}






































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipGeomDeviceComp27(real* DD, 
												 int* k_Q, 
												 real* QQ,
												 unsigned int sizeQ,
												 real om1, 
												 real* NormalX,
												 real* NormalY,
												 real* NormalZ,
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[dirE   *sizeQ];
      nx_dirW   = &NormalX[dirW   *sizeQ];
      nx_dirN   = &NormalX[dirN   *sizeQ];
      nx_dirS   = &NormalX[dirS   *sizeQ];
      nx_dirT   = &NormalX[dirT   *sizeQ];
      nx_dirB   = &NormalX[dirB   *sizeQ];
      nx_dirNE  = &NormalX[dirNE  *sizeQ];
      nx_dirSW  = &NormalX[dirSW  *sizeQ];
      nx_dirSE  = &NormalX[dirSE  *sizeQ];
      nx_dirNW  = &NormalX[dirNW  *sizeQ];
      nx_dirTE  = &NormalX[dirTE  *sizeQ];
      nx_dirBW  = &NormalX[dirBW  *sizeQ];
      nx_dirBE  = &NormalX[dirBE  *sizeQ];
      nx_dirTW  = &NormalX[dirTW  *sizeQ];
      nx_dirTN  = &NormalX[dirTN  *sizeQ];
      nx_dirBS  = &NormalX[dirBS  *sizeQ];
      nx_dirBN  = &NormalX[dirBN  *sizeQ];
      nx_dirTS  = &NormalX[dirTS  *sizeQ];
      nx_dirTNE = &NormalX[dirTNE *sizeQ];
      nx_dirTSW = &NormalX[dirTSW *sizeQ];
      nx_dirTSE = &NormalX[dirTSE *sizeQ];
      nx_dirTNW = &NormalX[dirTNW *sizeQ];
      nx_dirBNE = &NormalX[dirBNE *sizeQ];
      nx_dirBSW = &NormalX[dirBSW *sizeQ];
      nx_dirBSE = &NormalX[dirBSE *sizeQ];
      nx_dirBNW = &NormalX[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[dirE   *sizeQ];
      ny_dirW   = &NormalY[dirW   *sizeQ];
      ny_dirN   = &NormalY[dirN   *sizeQ];
      ny_dirS   = &NormalY[dirS   *sizeQ];
      ny_dirT   = &NormalY[dirT   *sizeQ];
      ny_dirB   = &NormalY[dirB   *sizeQ];
      ny_dirNE  = &NormalY[dirNE  *sizeQ];
      ny_dirSW  = &NormalY[dirSW  *sizeQ];
      ny_dirSE  = &NormalY[dirSE  *sizeQ];
      ny_dirNW  = &NormalY[dirNW  *sizeQ];
      ny_dirTE  = &NormalY[dirTE  *sizeQ];
      ny_dirBW  = &NormalY[dirBW  *sizeQ];
      ny_dirBE  = &NormalY[dirBE  *sizeQ];
      ny_dirTW  = &NormalY[dirTW  *sizeQ];
      ny_dirTN  = &NormalY[dirTN  *sizeQ];
      ny_dirBS  = &NormalY[dirBS  *sizeQ];
      ny_dirBN  = &NormalY[dirBN  *sizeQ];
      ny_dirTS  = &NormalY[dirTS  *sizeQ];
      ny_dirTNE = &NormalY[dirTNE *sizeQ];
      ny_dirTSW = &NormalY[dirTSW *sizeQ];
      ny_dirTSE = &NormalY[dirTSE *sizeQ];
      ny_dirTNW = &NormalY[dirTNW *sizeQ];
      ny_dirBNE = &NormalY[dirBNE *sizeQ];
      ny_dirBSW = &NormalY[dirBSW *sizeQ];
      ny_dirBSE = &NormalY[dirBSE *sizeQ];
      ny_dirBNW = &NormalY[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[dirE   *sizeQ];
      nz_dirW   = &NormalZ[dirW   *sizeQ];
      nz_dirN   = &NormalZ[dirN   *sizeQ];
      nz_dirS   = &NormalZ[dirS   *sizeQ];
      nz_dirT   = &NormalZ[dirT   *sizeQ];
      nz_dirB   = &NormalZ[dirB   *sizeQ];
      nz_dirNE  = &NormalZ[dirNE  *sizeQ];
      nz_dirSW  = &NormalZ[dirSW  *sizeQ];
      nz_dirSE  = &NormalZ[dirSE  *sizeQ];
      nz_dirNW  = &NormalZ[dirNW  *sizeQ];
      nz_dirTE  = &NormalZ[dirTE  *sizeQ];
      nz_dirBW  = &NormalZ[dirBW  *sizeQ];
      nz_dirBE  = &NormalZ[dirBE  *sizeQ];
      nz_dirTW  = &NormalZ[dirTW  *sizeQ];
      nz_dirTN  = &NormalZ[dirTN  *sizeQ];
      nz_dirBS  = &NormalZ[dirBS  *sizeQ];
      nz_dirBN  = &NormalZ[dirBN  *sizeQ];
      nz_dirTS  = &NormalZ[dirTS  *sizeQ];
      nz_dirTNE = &NormalZ[dirTNE *sizeQ];
      nz_dirTSW = &NormalZ[dirTSW *sizeQ];
      nz_dirTSE = &NormalZ[dirTSE *sizeQ];
      nz_dirTNW = &NormalZ[dirTNW *sizeQ];
      nz_dirBNE = &NormalZ[dirBNE *sizeQ];
      nz_dirBSW = &NormalZ[dirBSW *sizeQ];
      nz_dirBSE = &NormalZ[dirBSE *sizeQ];
      nz_dirBNW = &NormalZ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[dirE   ])[ke   ];
      real f_E    = (D.f[dirW   ])[kw   ];
      real f_S    = (D.f[dirN   ])[kn   ];
      real f_N    = (D.f[dirS   ])[ks   ];
      real f_B    = (D.f[dirT   ])[kt   ];
      real f_T    = (D.f[dirB   ])[kb   ];
      real f_SW   = (D.f[dirNE  ])[kne  ];
      real f_NE   = (D.f[dirSW  ])[ksw  ];
      real f_NW   = (D.f[dirSE  ])[kse  ];
      real f_SE   = (D.f[dirNW  ])[knw  ];
      real f_BW   = (D.f[dirTE  ])[kte  ];
      real f_TE   = (D.f[dirBW  ])[kbw  ];
      real f_TW   = (D.f[dirBE  ])[kbe  ];
      real f_BE   = (D.f[dirTW  ])[ktw  ];
      real f_BS   = (D.f[dirTN  ])[ktn  ];
      real f_TN   = (D.f[dirBS  ])[kbs  ];
      real f_TS   = (D.f[dirBN  ])[kbn  ];
      real f_BN   = (D.f[dirTS  ])[kts  ];
      real f_BSW  = (D.f[dirTNE ])[ktne ];
      real f_BNE  = (D.f[dirTSW ])[ktsw ];
      real f_BNW  = (D.f[dirTSE ])[ktse ];
      real f_BSE  = (D.f[dirTNW ])[ktnw ];
      real f_TSW  = (D.f[dirBNE ])[kbne ];
      real f_TNE  = (D.f[dirBSW ])[kbsw ];
      real f_TNW  = (D.f[dirBSE ])[kbse ];
      real f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real VeloX = vx1;
	  real VeloY = vx2;
	  real VeloZ = vx3;
	  real fac = c0o1;//0.5;
 	  real phi = c0o1;
	  //real alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real kxyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho) - ((vx1*vx2)));
      real kyzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho) - ((vx2*vx3)));
      real kxzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho) - ((vx1*vx3)));

	  real kxxFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx1*vx1)));
	  real kyyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx2*vx2)));
	  real kzzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx3*vx3)));

	  real magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

	  //fac = fac * magS / (c1o3 * (one / om1 - c1o2));
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real *facAst = &QQ[dirZERO *sizeQ];

	  //fac = fac * alpha + facAst[k] * (one - alpha);
	  //facAst[k] = fac;
	  //(&QQ[dirZERO *sizeQ])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////real uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real phi = expf(magS/0.01f) - one;
	  //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real C = five;
	  //real kappa = 0.41f;
	  //real phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
	  //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
	  //real qMed = sum/count;
	  //real phi = fac / (qMed + fac);
	  //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real testQ = c2o1;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirE[k]) + fac);
		 VeloX *= phi;
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirW[k]) + fac);
		 VeloX *= phi;
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirN[k]) + fac);
		 VeloY *= phi;
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirS[k]) + fac);
		 VeloY *= phi;
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nz_dirT[k]) + fac);
		 VeloZ *= phi;
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nz_dirB[k]) + fac);
		 VeloZ *= phi;
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
		 VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirNE[k] + ny_dirNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
		 VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirSW[k] - ny_dirSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
		 VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirSE[k] - ny_dirSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
		 VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirNW[k] + ny_dirNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTE[k] + nz_dirTE[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBW[k] - nz_dirBW[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBE[k] - nz_dirBE[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTW[k] + nz_dirTW[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
		 VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirTN[k] + nz_dirTN[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
		 VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirBS[k] - nz_dirBS[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
		 VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirBN[k] - nz_dirBN[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
		 VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirTS[k] + nz_dirTS[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
		 VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
		 VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
		 VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
		 VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
		 VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
		 VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
		 VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = (phi > one) ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
		 VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipNormDeviceComp27(real* DD, 
												 int* k_Q, 
												 real* QQ,
												 unsigned int sizeQ,
												 real om1, 
												 real* NormalX,
												 real* NormalY,
												 real* NormalZ,
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[dirE   *sizeQ];
      nx_dirW   = &NormalX[dirW   *sizeQ];
      nx_dirN   = &NormalX[dirN   *sizeQ];
      nx_dirS   = &NormalX[dirS   *sizeQ];
      nx_dirT   = &NormalX[dirT   *sizeQ];
      nx_dirB   = &NormalX[dirB   *sizeQ];
      nx_dirNE  = &NormalX[dirNE  *sizeQ];
      nx_dirSW  = &NormalX[dirSW  *sizeQ];
      nx_dirSE  = &NormalX[dirSE  *sizeQ];
      nx_dirNW  = &NormalX[dirNW  *sizeQ];
      nx_dirTE  = &NormalX[dirTE  *sizeQ];
      nx_dirBW  = &NormalX[dirBW  *sizeQ];
      nx_dirBE  = &NormalX[dirBE  *sizeQ];
      nx_dirTW  = &NormalX[dirTW  *sizeQ];
      nx_dirTN  = &NormalX[dirTN  *sizeQ];
      nx_dirBS  = &NormalX[dirBS  *sizeQ];
      nx_dirBN  = &NormalX[dirBN  *sizeQ];
      nx_dirTS  = &NormalX[dirTS  *sizeQ];
      nx_dirTNE = &NormalX[dirTNE *sizeQ];
      nx_dirTSW = &NormalX[dirTSW *sizeQ];
      nx_dirTSE = &NormalX[dirTSE *sizeQ];
      nx_dirTNW = &NormalX[dirTNW *sizeQ];
      nx_dirBNE = &NormalX[dirBNE *sizeQ];
      nx_dirBSW = &NormalX[dirBSW *sizeQ];
      nx_dirBSE = &NormalX[dirBSE *sizeQ];
      nx_dirBNW = &NormalX[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[dirE   *sizeQ];
      ny_dirW   = &NormalY[dirW   *sizeQ];
      ny_dirN   = &NormalY[dirN   *sizeQ];
      ny_dirS   = &NormalY[dirS   *sizeQ];
      ny_dirT   = &NormalY[dirT   *sizeQ];
      ny_dirB   = &NormalY[dirB   *sizeQ];
      ny_dirNE  = &NormalY[dirNE  *sizeQ];
      ny_dirSW  = &NormalY[dirSW  *sizeQ];
      ny_dirSE  = &NormalY[dirSE  *sizeQ];
      ny_dirNW  = &NormalY[dirNW  *sizeQ];
      ny_dirTE  = &NormalY[dirTE  *sizeQ];
      ny_dirBW  = &NormalY[dirBW  *sizeQ];
      ny_dirBE  = &NormalY[dirBE  *sizeQ];
      ny_dirTW  = &NormalY[dirTW  *sizeQ];
      ny_dirTN  = &NormalY[dirTN  *sizeQ];
      ny_dirBS  = &NormalY[dirBS  *sizeQ];
      ny_dirBN  = &NormalY[dirBN  *sizeQ];
      ny_dirTS  = &NormalY[dirTS  *sizeQ];
      ny_dirTNE = &NormalY[dirTNE *sizeQ];
      ny_dirTSW = &NormalY[dirTSW *sizeQ];
      ny_dirTSE = &NormalY[dirTSE *sizeQ];
      ny_dirTNW = &NormalY[dirTNW *sizeQ];
      ny_dirBNE = &NormalY[dirBNE *sizeQ];
      ny_dirBSW = &NormalY[dirBSW *sizeQ];
      ny_dirBSE = &NormalY[dirBSE *sizeQ];
      ny_dirBNW = &NormalY[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[dirE   *sizeQ];
      nz_dirW   = &NormalZ[dirW   *sizeQ];
      nz_dirN   = &NormalZ[dirN   *sizeQ];
      nz_dirS   = &NormalZ[dirS   *sizeQ];
      nz_dirT   = &NormalZ[dirT   *sizeQ];
      nz_dirB   = &NormalZ[dirB   *sizeQ];
      nz_dirNE  = &NormalZ[dirNE  *sizeQ];
      nz_dirSW  = &NormalZ[dirSW  *sizeQ];
      nz_dirSE  = &NormalZ[dirSE  *sizeQ];
      nz_dirNW  = &NormalZ[dirNW  *sizeQ];
      nz_dirTE  = &NormalZ[dirTE  *sizeQ];
      nz_dirBW  = &NormalZ[dirBW  *sizeQ];
      nz_dirBE  = &NormalZ[dirBE  *sizeQ];
      nz_dirTW  = &NormalZ[dirTW  *sizeQ];
      nz_dirTN  = &NormalZ[dirTN  *sizeQ];
      nz_dirBS  = &NormalZ[dirBS  *sizeQ];
      nz_dirBN  = &NormalZ[dirBN  *sizeQ];
      nz_dirTS  = &NormalZ[dirTS  *sizeQ];
      nz_dirTNE = &NormalZ[dirTNE *sizeQ];
      nz_dirTSW = &NormalZ[dirTSW *sizeQ];
      nz_dirTSE = &NormalZ[dirTSE *sizeQ];
      nz_dirTNW = &NormalZ[dirTNW *sizeQ];
      nz_dirBNE = &NormalZ[dirBNE *sizeQ];
      nz_dirBSW = &NormalZ[dirBSW *sizeQ];
      nz_dirBSE = &NormalZ[dirBSE *sizeQ];
      nz_dirBNW = &NormalZ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[dirE   ])[ke   ];
      real f_E    = (D.f[dirW   ])[kw   ];
      real f_S    = (D.f[dirN   ])[kn   ];
      real f_N    = (D.f[dirS   ])[ks   ];
      real f_B    = (D.f[dirT   ])[kt   ];
      real f_T    = (D.f[dirB   ])[kb   ];
      real f_SW   = (D.f[dirNE  ])[kne  ];
      real f_NE   = (D.f[dirSW  ])[ksw  ];
      real f_NW   = (D.f[dirSE  ])[kse  ];
      real f_SE   = (D.f[dirNW  ])[knw  ];
      real f_BW   = (D.f[dirTE  ])[kte  ];
      real f_TE   = (D.f[dirBW  ])[kbw  ];
      real f_TW   = (D.f[dirBE  ])[kbe  ];
      real f_BE   = (D.f[dirTW  ])[ktw  ];
      real f_BS   = (D.f[dirTN  ])[ktn  ];
      real f_TN   = (D.f[dirBS  ])[kbs  ];
      real f_TS   = (D.f[dirBN  ])[kbn  ];
      real f_BN   = (D.f[dirTS  ])[kts  ];
      real f_BSW  = (D.f[dirTNE ])[ktne ];
      real f_BNE  = (D.f[dirTSW ])[ktsw ];
      real f_BNW  = (D.f[dirTSE ])[ktse ];
      real f_BSE  = (D.f[dirTNW ])[ktnw ];
      real f_TSW  = (D.f[dirBNE ])[kbne ];
      real f_TNE  = (D.f[dirBSW ])[kbsw ];
      real f_TNW  = (D.f[dirBSE ])[kbse ];
      real f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real VeloX = vx1;
	  real VeloY = vx2;
	  real VeloZ = vx3;
	  real fac = c1o100;//0.5;
 	  //real phi = c0o1;
	  real alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real kxyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho) - ((vx1*vx2)));
      real kyzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho) - ((vx2*vx3)));
      real kxzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho) - ((vx1*vx3)));

	  real kxxFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx1*vx1)));
	  real kyyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx2*vx2)));
	  real kzzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx3*vx3)));

	  real magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

	  fac = fac * magS / (c1o3 * (c1o1 / om1 - c1o2));
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real *facAst = &QQ[dirZERO *sizeQ];

	  fac = fac * alpha + facAst[k] * (c1o1 - alpha);
	  facAst[k] = fac;
	  //(&QQ[dirZERO *sizeQ])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////real uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real phi = expf(magS/0.01f) - one;
	  //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real C = five;
	  //real kappa = 0.41f;
	  //real phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
	  //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
	  //real qMed = sum/count;
	  //real phi = fac / (qMed + fac);
	  //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real sliplength = 0.9f;//c1o2;
	  real qSlip = c0o1;
	  real un = c0o1;
	  real ut = c0o1;
	  real tangential = c0o1;
	  //real smallSingle = Op0000002;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
		 un = fabs((vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k]);
		 ut = fabs(VeloX);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
		 un = fabs(-(vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k]);
		 ut = fabs(-VeloX);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
		 un = fabs( (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k]);
		 ut = fabs( VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
		 un = fabs(-(vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k]);
		 ut = fabs(-VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
		 un = fabs( (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k]);
		 ut = fabs( VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nz_dirT[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
		 un = fabs(-(vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k]);
		 ut = fabs(-VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nz_dirB[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
		 VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
		 un = fabs( (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k] + (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k]);
		 ut = fabs( VeloX + VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirNE[k]+ny_dirNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
		 VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
		 un = fabs(-(vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k] - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k]);
		 ut = fabs(-VeloX - VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirSW[k]-ny_dirSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
		 VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
		 un = fabs( (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k] - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k]);
		 ut = fabs( VeloX - VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirSE[k]-ny_dirSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
		 VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
		 un = fabs(-(vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k] + (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k]);
		 ut = fabs(-VeloX + VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirNW[k]+ny_dirNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
		 un = fabs( (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k] + (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k]);
		 ut = fabs( VeloX + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTE[k]+nz_dirTE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
		 un = fabs(-(vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k] - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k]);
		 ut = fabs(-VeloX - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBW[k]-nz_dirBW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
		 un = fabs( (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k] - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k]);
		 ut = fabs( VeloX - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBE[k]-nz_dirBE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
		 un = fabs(-(vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k] + (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k]);
		 ut = fabs(-VeloX + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTW[k]+nz_dirTW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
		 VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
		 un = fabs( (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k] + (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k]);
		 ut = fabs( VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirTN[k]+nz_dirTN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
		 VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
		 un = fabs(-(vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k] - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k]);
		 ut = fabs(-VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirBS[k]-nz_dirBS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
		 VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
		 un = fabs( (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k] - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k]);
		 ut = fabs( VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirBN[k]-nz_dirBN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
		 VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
		 un = fabs(-(vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k] + (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k]);
		 ut = fabs(-VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirTS[k]+nz_dirTS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
		 VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
		 un = fabs( (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k] 
				   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k] 
				   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k]);
		 ut = fabs( VeloX + VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
		 VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
		 un = fabs(-(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k] 
				   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k] 
				   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k]);
		 ut = fabs(-VeloX - VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
		 VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
		 un = fabs( (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k] 
				   +(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k] 
				   -(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k]);
		 ut = fabs( VeloX + VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
		 VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
		 un = fabs(-(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k] 
				   -(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k] 
				   +(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k]);
		 ut = fabs(-VeloX - VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
		 VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
		 un = fabs(+(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k] 
				   -(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k] 
				   +(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k]);
		 ut = fabs(+VeloX - VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
		 VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
		 un = fabs(-(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k] 
				   +(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k] 
				   -(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k]);
		 ut = fabs(-VeloX + VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
		 VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
		 un = fabs( (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k] 
				   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k] 
				   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k]);
		 ut = fabs( VeloX - VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
		 VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
		 un = fabs(-(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k] 
				   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k] 
				   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k]);
		 ut = fabs(-VeloX + VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q) - c1o216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


