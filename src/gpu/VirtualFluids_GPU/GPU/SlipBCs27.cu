#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include "lbm/constants/NumericConstants.h"
#include "KernelUtilities.h"

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;

//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDevice27(real* DD, 
                                         int* k_Q, 
                                         real* QQ,
                                         unsigned int numberOfBCnodes,
                                         real om1, 
                                         unsigned int* neighborX,
                                         unsigned int* neighborY,
                                         unsigned int* neighborZ,
                                         unsigned int size_Mat, 
                                         bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[E   ] = &DD[E   *size_Mat];
      D.f[W   ] = &DD[W   *size_Mat];
      D.f[N   ] = &DD[N   *size_Mat];
      D.f[S   ] = &DD[S   *size_Mat];
      D.f[T   ] = &DD[T   *size_Mat];
      D.f[B   ] = &DD[B   *size_Mat];
      D.f[NE  ] = &DD[NE  *size_Mat];
      D.f[SW  ] = &DD[SW  *size_Mat];
      D.f[SE  ] = &DD[SE  *size_Mat];
      D.f[NW  ] = &DD[NW  *size_Mat];
      D.f[TE  ] = &DD[TE  *size_Mat];
      D.f[BW  ] = &DD[BW  *size_Mat];
      D.f[BE  ] = &DD[BE  *size_Mat];
      D.f[TW  ] = &DD[TW  *size_Mat];
      D.f[TN  ] = &DD[TN  *size_Mat];
      D.f[BS  ] = &DD[BS  *size_Mat];
      D.f[BN  ] = &DD[BN  *size_Mat];
      D.f[TS  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[TNE *size_Mat];
      D.f[TSW ] = &DD[TSW *size_Mat];
      D.f[TSE ] = &DD[TSE *size_Mat];
      D.f[TNW ] = &DD[TNW *size_Mat];
      D.f[BNE ] = &DD[BNE *size_Mat];
      D.f[BSW ] = &DD[BSW *size_Mat];
      D.f[BSE ] = &DD[BSE *size_Mat];
      D.f[BNW ] = &DD[BNW *size_Mat];
   } 
   else
   {
      D.f[W   ] = &DD[E   *size_Mat];
      D.f[E   ] = &DD[W   *size_Mat];
      D.f[S   ] = &DD[N   *size_Mat];
      D.f[N   ] = &DD[S   *size_Mat];
      D.f[B   ] = &DD[T   *size_Mat];
      D.f[T   ] = &DD[B   *size_Mat];
      D.f[SW  ] = &DD[NE  *size_Mat];
      D.f[NE  ] = &DD[SW  *size_Mat];
      D.f[NW  ] = &DD[SE  *size_Mat];
      D.f[SE  ] = &DD[NW  *size_Mat];
      D.f[BW  ] = &DD[TE  *size_Mat];
      D.f[TE  ] = &DD[BW  *size_Mat];
      D.f[TW  ] = &DD[BE  *size_Mat];
      D.f[BE  ] = &DD[TW  *size_Mat];
      D.f[BS  ] = &DD[TN  *size_Mat];
      D.f[TN  ] = &DD[BS  *size_Mat];
      D.f[TS  ] = &DD[BN  *size_Mat];
      D.f[BN  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[BSW *size_Mat];
      D.f[TSW ] = &DD[BNE *size_Mat];
      D.f[TSE ] = &DD[BNW *size_Mat];
      D.f[TNW ] = &DD[BSE *size_Mat];
      D.f[BNE ] = &DD[TSW *size_Mat];
      D.f[BSW ] = &DD[TNE *size_Mat];
      D.f[BSE ] = &DD[TNW *size_Mat];
      D.f[BNW ] = &DD[TSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[E   * numberOfBCnodes];
      q_dirW   = &QQ[W   * numberOfBCnodes];
      q_dirN   = &QQ[N   * numberOfBCnodes];
      q_dirS   = &QQ[S   * numberOfBCnodes];
      q_dirT   = &QQ[T   * numberOfBCnodes];
      q_dirB   = &QQ[B   * numberOfBCnodes];
      q_dirNE  = &QQ[NE  * numberOfBCnodes];
      q_dirSW  = &QQ[SW  * numberOfBCnodes];
      q_dirSE  = &QQ[SE  * numberOfBCnodes];
      q_dirNW  = &QQ[NW  * numberOfBCnodes];
      q_dirTE  = &QQ[TE  * numberOfBCnodes];
      q_dirBW  = &QQ[BW  * numberOfBCnodes];
      q_dirBE  = &QQ[BE  * numberOfBCnodes];
      q_dirTW  = &QQ[TW  * numberOfBCnodes];
      q_dirTN  = &QQ[TN  * numberOfBCnodes];
      q_dirBS  = &QQ[BS  * numberOfBCnodes];
      q_dirBN  = &QQ[BN  * numberOfBCnodes];
      q_dirTS  = &QQ[TS  * numberOfBCnodes];
      q_dirTNE = &QQ[TNE * numberOfBCnodes];
      q_dirTSW = &QQ[TSW * numberOfBCnodes];
      q_dirTSE = &QQ[TSE * numberOfBCnodes];
      q_dirTNW = &QQ[TNW * numberOfBCnodes];
      q_dirBNE = &QQ[BNE * numberOfBCnodes];
      q_dirBSW = &QQ[BSW * numberOfBCnodes];
      q_dirBSE = &QQ[BSE * numberOfBCnodes];
      q_dirBNW = &QQ[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[E   ])[ke   ];
      real f_E    = (D.f[W   ])[kw   ];
      real f_S    = (D.f[N   ])[kn   ];
      real f_N    = (D.f[S   ])[ks   ];
      real f_B    = (D.f[T   ])[kt   ];
      real f_T    = (D.f[B   ])[kb   ];
      real f_SW   = (D.f[NE  ])[kne  ];
      real f_NE   = (D.f[SW  ])[ksw  ];
      real f_NW   = (D.f[SE  ])[kse  ];
      real f_SE   = (D.f[NW  ])[knw  ];
      real f_BW   = (D.f[TE  ])[kte  ];
      real f_TE   = (D.f[BW  ])[kbw  ];
      real f_TW   = (D.f[BE  ])[kbe  ];
      real f_BE   = (D.f[TW  ])[ktw  ];
      real f_BS   = (D.f[TN  ])[ktn  ];
      real f_TN   = (D.f[BS  ])[kbs  ];
      real f_TS   = (D.f[BN  ])[kbn  ];
      real f_BN   = (D.f[TS  ])[kts  ];
      real f_BSW  = (D.f[TNE ])[ktne ];
      real f_BNE  = (D.f[TSW ])[ktsw ];
      real f_BNW  = (D.f[TSE ])[ktse ];
      real f_BSE  = (D.f[TNW ])[ktnw ];
      real f_TSW  = (D.f[BNE ])[kbne ];
      real f_TNE  = (D.f[BSW ])[kbsw ];
      real f_TNW  = (D.f[BSE ])[kbse ];
      real f_TSE  = (D.f[BNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[REST])[kzero]); 

      vx1    =  ((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W); 
         

      vx2    =   (-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S); 

      vx3    =   ((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[E   ] = &DD[E   *size_Mat];
         D.f[W   ] = &DD[W   *size_Mat];
         D.f[N   ] = &DD[N   *size_Mat];
         D.f[S   ] = &DD[S   *size_Mat];
         D.f[T   ] = &DD[T   *size_Mat];
         D.f[B   ] = &DD[B   *size_Mat];
         D.f[NE  ] = &DD[NE  *size_Mat];
         D.f[SW  ] = &DD[SW  *size_Mat];
         D.f[SE  ] = &DD[SE  *size_Mat];
         D.f[NW  ] = &DD[NW  *size_Mat];
         D.f[TE  ] = &DD[TE  *size_Mat];
         D.f[BW  ] = &DD[BW  *size_Mat];
         D.f[BE  ] = &DD[BE  *size_Mat];
         D.f[TW  ] = &DD[TW  *size_Mat];
         D.f[TN  ] = &DD[TN  *size_Mat];
         D.f[BS  ] = &DD[BS  *size_Mat];
         D.f[BN  ] = &DD[BN  *size_Mat];
         D.f[TS  ] = &DD[TS  *size_Mat];
         D.f[REST] = &DD[REST*size_Mat];
         D.f[TNE ] = &DD[TNE *size_Mat];
         D.f[TSW ] = &DD[TSW *size_Mat];
         D.f[TSE ] = &DD[TSE *size_Mat];
         D.f[TNW ] = &DD[TNW *size_Mat];
         D.f[BNE ] = &DD[BNE *size_Mat];
         D.f[BSW ] = &DD[BSW *size_Mat];
         D.f[BSE ] = &DD[BSE *size_Mat];
         D.f[BNW ] = &DD[BNW *size_Mat];
      } 
      else
      {
         D.f[W   ] = &DD[E   *size_Mat];
         D.f[E   ] = &DD[W   *size_Mat];
         D.f[S   ] = &DD[N   *size_Mat];
         D.f[N   ] = &DD[S   *size_Mat];
         D.f[B   ] = &DD[T   *size_Mat];
         D.f[T   ] = &DD[B   *size_Mat];
         D.f[SW  ] = &DD[NE  *size_Mat];
         D.f[NE  ] = &DD[SW  *size_Mat];
         D.f[NW  ] = &DD[SE  *size_Mat];
         D.f[SE  ] = &DD[NW  *size_Mat];
         D.f[BW  ] = &DD[TE  *size_Mat];
         D.f[TE  ] = &DD[BW  *size_Mat];
         D.f[TW  ] = &DD[BE  *size_Mat];
         D.f[BE  ] = &DD[TW  *size_Mat];
         D.f[BS  ] = &DD[TN  *size_Mat];
         D.f[TN  ] = &DD[BS  *size_Mat];
         D.f[TS  ] = &DD[BN  *size_Mat];
         D.f[BN  ] = &DD[TS  *size_Mat];
         D.f[REST] = &DD[REST*size_Mat];
         D.f[TNE ] = &DD[BSW *size_Mat];
         D.f[TSW ] = &DD[BNE *size_Mat];
         D.f[TSE ] = &DD[BNW *size_Mat];
         D.f[TNW ] = &DD[BSE *size_Mat];
         D.f[BNE ] = &DD[TSW *size_Mat];
         D.f[BSW ] = &DD[TNE *size_Mat];
         D.f[BSE ] = &DD[TNW *size_Mat];
         D.f[BNW ] = &DD[TSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[REST])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real fac = c1o1;//c99o100;
	  real VeloX = fac*vx1;
	  real VeloY = fac*vx2;
	  real VeloZ = fac*vx3;
	  bool x = false;
	  bool y = false;
	  bool z = false;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho+c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq); 
         (D.f[W])[kw]=(c1o1-q)/(c1o1+q)*(f_E-feq*om1)/(c1o1-om1)+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q);
         //(D.f[W])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho+c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq); 
         (D.f[E])[ke]=(c1o1-q)/(c1o1+q)*(f_W-feq*om1)/(c1o1-om1)+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q);
         //(D.f[E])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho+c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq); 
         (D.f[S])[ks]=(c1o1-q)/(c1o1+q)*(f_N-feq*om1)/(c1o1-om1)+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q);
         //(D.f[S])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho+c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq); 
         (D.f[N])[kn]=(c1o1-q)/(c1o1+q)*(f_S-feq*om1)/(c1o1-om1)+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q);
         //(D.f[N])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho+c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq); 
         (D.f[B])[kb]=(c1o1-q)/(c1o1+q)*(f_T-feq*om1)/(c1o1-om1)+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q);
         //(D.f[B])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho+c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq); 
         (D.f[T])[kt]=(c1o1-q)/(c1o1+q)*(f_B-feq*om1)/(c1o1-om1)+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q);
         //(D.f[T])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         (D.f[SW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-feq*om1)/(c1o1-om1)+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
         //(D.f[SW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         (D.f[NE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-feq*om1)/(c1o1-om1)+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
         //(D.f[NE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         (D.f[NW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-feq*om1)/(c1o1-om1)+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
         //(D.f[NW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         (D.f[SE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-feq*om1)/(c1o1-om1)+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
         //(D.f[SE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         (D.f[BW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-feq*om1)/(c1o1-om1)+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
         //(D.f[BW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         (D.f[TE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-feq*om1)/(c1o1-om1)+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
         //(D.f[TE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         (D.f[TW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-feq*om1)/(c1o1-om1)+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
         //(D.f[TW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         (D.f[BE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-feq*om1)/(c1o1-om1)+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
         //(D.f[BE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         (D.f[BS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-feq*om1)/(c1o1-om1)+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
         //(D.f[BS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         (D.f[TN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-feq*om1)/(c1o1-om1)+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
         //(D.f[TN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         (D.f[TS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-feq*om1)/(c1o1-om1)+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
         //(D.f[TS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         (D.f[BN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-feq*om1)/(c1o1-om1)+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
         //(D.f[BN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         (D.f[BSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-feq*om1)/(c1o1-om1)+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
         //(D.f[BSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         (D.f[TNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-feq*om1)/(c1o1-om1)+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
         //(D.f[TNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         (D.f[TSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-feq*om1)/(c1o1-om1)+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
         //(D.f[TSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         (D.f[BNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-feq*om1)/(c1o1-om1)+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
         //(D.f[BNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         (D.f[BNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-feq*om1)/(c1o1-om1)+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
         //(D.f[BNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         (D.f[TSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-feq*om1)/(c1o1-om1)+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
         //(D.f[TSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         (D.f[TNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-feq*om1)/(c1o1-om1)+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
         //(D.f[TNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         (D.f[BSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-feq*om1)/(c1o1-om1)+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
         //(D.f[BSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDeviceComp27(
                                    real* distributions, 
                                    int* subgridDistanceIndices, 
                                    real* subgridDistances,
                                    unsigned int numberOfBCnodes,
                                    real omega, 
                                    unsigned int* neighborX,
                                    unsigned int* neighborY,
                                    unsigned int* neighborZ,
                                    unsigned int numberOfLBnodes, 
                                    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!
   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned  x = threadIdx.x;  // global x-index 
   const unsigned  y = blockIdx.x;   // global y-index 
   const unsigned  z = blockIdx.y;   // global z-index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;

   if(k < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[k];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[E   ])[ke   ];
      real f_E    = (dist.f[W   ])[kw   ];
      real f_S    = (dist.f[N   ])[kn   ];
      real f_N    = (dist.f[S   ])[ks   ];
      real f_B    = (dist.f[T   ])[kt   ];
      real f_T    = (dist.f[B   ])[kb   ];
      real f_SW   = (dist.f[NE  ])[kne  ];
      real f_NE   = (dist.f[SW  ])[ksw  ];
      real f_NW   = (dist.f[SE  ])[kse  ];
      real f_SE   = (dist.f[NW  ])[knw  ];
      real f_BW   = (dist.f[TE  ])[kte  ];
      real f_TE   = (dist.f[BW  ])[kbw  ];
      real f_TW   = (dist.f[BE  ])[kbe  ];
      real f_BE   = (dist.f[TW  ])[ktw  ];
      real f_BS   = (dist.f[TN  ])[ktn  ];
      real f_TN   = (dist.f[BS  ])[kbs  ];
      real f_TS   = (dist.f[BN  ])[kbn  ];
      real f_BN   = (dist.f[TS  ])[kts  ];
      real f_BSW  = (dist.f[TNE ])[ktne ];
      real f_BNE  = (dist.f[TSW ])[ktsw ];
      real f_BNW  = (dist.f[TSE ])[ktse ];
      real f_BSE  = (dist.f[TNW ])[ktnw ];
      real f_TSW  = (dist.f[BNE ])[kbne ];
      real f_TNE  = (dist.f[BSW ])[kbsw ];
      real f_TNW  = (dist.f[BSE ])[kbse ];
      real f_TSE  = (dist.f[BNW ])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[REST])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real feq, q, velocityLB, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[E])[k];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloX;
         (dist.f[W])[kw] = getInterpolatedDistributionForVeloBC(q, f_E, f_W, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[W])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloX;
         (dist.f[E])[ke] = getInterpolatedDistributionForVeloBC(q, f_W, f_E, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[N])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloY;
         (dist.f[S])[ks] = getInterpolatedDistributionForVeloBC(q, f_N, f_S, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[S])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloY;
         (dist.f[N])[kn] = getInterpolatedDistributionForVeloBC(q, f_S, f_N, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[T])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloZ;
         (dist.f[B])[kb] = getInterpolatedDistributionForVeloBC(q, f_T, f_B, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[B])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloZ;
         (dist.f[T])[kt] = getInterpolatedDistributionForVeloBC(q, f_B, f_T, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[NE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloY;
         (dist.f[SW])[ksw] = getInterpolatedDistributionForVeloBC(q, f_NE, f_SW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[SW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloY;
         (dist.f[NE])[kne] = getInterpolatedDistributionForVeloBC(q, f_SW, f_NE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[SE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloY;
         (dist.f[NW])[knw] = getInterpolatedDistributionForVeloBC(q, f_SE, f_NW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[NW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloY;
         (dist.f[SE])[kse] = getInterpolatedDistributionForVeloBC(q, f_NW, f_SE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[TE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloZ;
         (dist.f[BW])[kbw] = getInterpolatedDistributionForVeloBC(q, f_TE, f_BW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[BW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloZ;
         (dist.f[TE])[kte] = getInterpolatedDistributionForVeloBC(q, f_BW, f_TE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[BE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloZ;
         (dist.f[TW])[ktw] = getInterpolatedDistributionForVeloBC(q, f_BE, f_TW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[TW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloZ;
         (dist.f[BE])[kbe] = getInterpolatedDistributionForVeloBC(q, f_TW, f_BE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[TN])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY + VeloZ;
         (dist.f[BS])[kbs] = getInterpolatedDistributionForVeloBC(q, f_TN, f_BS, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[BS])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY - VeloZ;
         (dist.f[TN])[ktn] = getInterpolatedDistributionForVeloBC(q, f_BS, f_TN, feq, omega, velocityBC, c1o54);
      }


      q = (subgridD.q[BN])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY - VeloZ;
         (dist.f[TS])[kts] = getInterpolatedDistributionForVeloBC(q, f_BN, f_TS, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[TS])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY + VeloZ;
         (dist.f[BN])[kbn] = getInterpolatedDistributionForVeloBC(q, f_TS, f_BN, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[TNE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[BSW])[kbsw] = getInterpolatedDistributionForVeloBC(q, f_TNE, f_BSW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[BSW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[TNE])[ktne] = getInterpolatedDistributionForVeloBC(q, f_BSW, f_TNE, feq, omega, velocityBC, c1o216);
      }


      q = (subgridD.q[BNE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[TSW])[ktsw] = getInterpolatedDistributionForVeloBC(q, f_BNE, f_TSW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[TSW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[BNE])[kbne] = getInterpolatedDistributionForVeloBC(q, f_TSW, f_BNE, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[TSE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[BNW])[kbnw] = getInterpolatedDistributionForVeloBC(q, f_TSE, f_BNW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[BNW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[TSE])[ktse] = getInterpolatedDistributionForVeloBC(q, f_BNW, f_TSE, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[BSE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[TNW])[ktnw] = getInterpolatedDistributionForVeloBC(q, f_BSE, f_TNW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[TNW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[BSE])[kbse] = getInterpolatedDistributionForVeloBC(q, f_TNW, f_BSE, feq, omega, velocityBC, c1o216);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDeviceComp27TurbViscosity(real* DD, 
											 int* k_Q, 
											 real* QQ,
											 unsigned int numberOfBCnodes,
											 real om1, 
											 unsigned int* neighborX,
											 unsigned int* neighborY,
											 unsigned int* neighborZ,
                                  real* turbViscosity,
											 unsigned int size_Mat, 
											 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[E   ] = &DD[E   *size_Mat];
      D.f[W   ] = &DD[W   *size_Mat];
      D.f[N   ] = &DD[N   *size_Mat];
      D.f[S   ] = &DD[S   *size_Mat];
      D.f[T   ] = &DD[T   *size_Mat];
      D.f[B   ] = &DD[B   *size_Mat];
      D.f[NE  ] = &DD[NE  *size_Mat];
      D.f[SW  ] = &DD[SW  *size_Mat];
      D.f[SE  ] = &DD[SE  *size_Mat];
      D.f[NW  ] = &DD[NW  *size_Mat];
      D.f[TE  ] = &DD[TE  *size_Mat];
      D.f[BW  ] = &DD[BW  *size_Mat];
      D.f[BE  ] = &DD[BE  *size_Mat];
      D.f[TW  ] = &DD[TW  *size_Mat];
      D.f[TN  ] = &DD[TN  *size_Mat];
      D.f[BS  ] = &DD[BS  *size_Mat];
      D.f[BN  ] = &DD[BN  *size_Mat];
      D.f[TS  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[TNE *size_Mat];
      D.f[TSW ] = &DD[TSW *size_Mat];
      D.f[TSE ] = &DD[TSE *size_Mat];
      D.f[TNW ] = &DD[TNW *size_Mat];
      D.f[BNE ] = &DD[BNE *size_Mat];
      D.f[BSW ] = &DD[BSW *size_Mat];
      D.f[BSE ] = &DD[BSE *size_Mat];
      D.f[BNW ] = &DD[BNW *size_Mat];
   } 
   else
   {
      D.f[W   ] = &DD[E   *size_Mat];
      D.f[E   ] = &DD[W   *size_Mat];
      D.f[S   ] = &DD[N   *size_Mat];
      D.f[N   ] = &DD[S   *size_Mat];
      D.f[B   ] = &DD[T   *size_Mat];
      D.f[T   ] = &DD[B   *size_Mat];
      D.f[SW  ] = &DD[NE  *size_Mat];
      D.f[NE  ] = &DD[SW  *size_Mat];
      D.f[NW  ] = &DD[SE  *size_Mat];
      D.f[SE  ] = &DD[NW  *size_Mat];
      D.f[BW  ] = &DD[TE  *size_Mat];
      D.f[TE  ] = &DD[BW  *size_Mat];
      D.f[TW  ] = &DD[BE  *size_Mat];
      D.f[BE  ] = &DD[TW  *size_Mat];
      D.f[BS  ] = &DD[TN  *size_Mat];
      D.f[TN  ] = &DD[BS  *size_Mat];
      D.f[TS  ] = &DD[BN  *size_Mat];
      D.f[BN  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[BSW *size_Mat];
      D.f[TSW ] = &DD[BNE *size_Mat];
      D.f[TSE ] = &DD[BNW *size_Mat];
      D.f[TNW ] = &DD[BSE *size_Mat];
      D.f[BNE ] = &DD[TSW *size_Mat];
      D.f[BSW ] = &DD[TNE *size_Mat];
      D.f[BSE ] = &DD[TNW *size_Mat];
      D.f[BNW ] = &DD[TSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[E   * numberOfBCnodes];
      q_dirW   = &QQ[W   * numberOfBCnodes];
      q_dirN   = &QQ[N   * numberOfBCnodes];
      q_dirS   = &QQ[S   * numberOfBCnodes];
      q_dirT   = &QQ[T   * numberOfBCnodes];
      q_dirB   = &QQ[B   * numberOfBCnodes];
      q_dirNE  = &QQ[NE  * numberOfBCnodes];
      q_dirSW  = &QQ[SW  * numberOfBCnodes];
      q_dirSE  = &QQ[SE  * numberOfBCnodes];
      q_dirNW  = &QQ[NW  * numberOfBCnodes];
      q_dirTE  = &QQ[TE  * numberOfBCnodes];
      q_dirBW  = &QQ[BW  * numberOfBCnodes];
      q_dirBE  = &QQ[BE  * numberOfBCnodes];
      q_dirTW  = &QQ[TW  * numberOfBCnodes];
      q_dirTN  = &QQ[TN  * numberOfBCnodes];
      q_dirBS  = &QQ[BS  * numberOfBCnodes];
      q_dirBN  = &QQ[BN  * numberOfBCnodes];
      q_dirTS  = &QQ[TS  * numberOfBCnodes];
      q_dirTNE = &QQ[TNE * numberOfBCnodes];
      q_dirTSW = &QQ[TSW * numberOfBCnodes];
      q_dirTSE = &QQ[TSE * numberOfBCnodes];
      q_dirTNW = &QQ[TNW * numberOfBCnodes];
      q_dirBNE = &QQ[BNE * numberOfBCnodes];
      q_dirBSW = &QQ[BSW * numberOfBCnodes];
      q_dirBSE = &QQ[BSE * numberOfBCnodes];
      q_dirBNW = &QQ[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[E   ])[ke   ];
      real f_E    = (D.f[W   ])[kw   ];
      real f_S    = (D.f[N   ])[kn   ];
      real f_N    = (D.f[S   ])[ks   ];
      real f_B    = (D.f[T   ])[kt   ];
      real f_T    = (D.f[B   ])[kb   ];
      real f_SW   = (D.f[NE  ])[kne  ];
      real f_NE   = (D.f[SW  ])[ksw  ];
      real f_NW   = (D.f[SE  ])[kse  ];
      real f_SE   = (D.f[NW  ])[knw  ];
      real f_BW   = (D.f[TE  ])[kte  ];
      real f_TE   = (D.f[BW  ])[kbw  ];
      real f_TW   = (D.f[BE  ])[kbe  ];
      real f_BE   = (D.f[TW  ])[ktw  ];
      real f_BS   = (D.f[TN  ])[ktn  ];
      real f_TN   = (D.f[BS  ])[kbs  ];
      real f_TS   = (D.f[BN  ])[kbn  ];
      real f_BN   = (D.f[TS  ])[kts  ];
      real f_BSW  = (D.f[TNE ])[ktne ];
      real f_BNE  = (D.f[TSW ])[ktsw ];
      real f_BNW  = (D.f[TSE ])[ktse ];
      real f_BSE  = (D.f[TNW ])[ktnw ];
      real f_TSW  = (D.f[BNE ])[kbne ];
      real f_TNE  = (D.f[BSW ])[kbsw ];
      real f_TNW  = (D.f[BSE ])[kbse ];
      real f_TSE  = (D.f[BNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[REST])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[E   ] = &DD[E   *size_Mat];
         D.f[W   ] = &DD[W   *size_Mat];
         D.f[N   ] = &DD[N   *size_Mat];
         D.f[S   ] = &DD[S   *size_Mat];
         D.f[T   ] = &DD[T   *size_Mat];
         D.f[B   ] = &DD[B   *size_Mat];
         D.f[NE  ] = &DD[NE  *size_Mat];
         D.f[SW  ] = &DD[SW  *size_Mat];
         D.f[SE  ] = &DD[SE  *size_Mat];
         D.f[NW  ] = &DD[NW  *size_Mat];
         D.f[TE  ] = &DD[TE  *size_Mat];
         D.f[BW  ] = &DD[BW  *size_Mat];
         D.f[BE  ] = &DD[BE  *size_Mat];
         D.f[TW  ] = &DD[TW  *size_Mat];
         D.f[TN  ] = &DD[TN  *size_Mat];
         D.f[BS  ] = &DD[BS  *size_Mat];
         D.f[BN  ] = &DD[BN  *size_Mat];
         D.f[TS  ] = &DD[TS  *size_Mat];
         D.f[REST] = &DD[REST*size_Mat];
         D.f[TNE ] = &DD[TNE *size_Mat];
         D.f[TSW ] = &DD[TSW *size_Mat];
         D.f[TSE ] = &DD[TSE *size_Mat];
         D.f[TNW ] = &DD[TNW *size_Mat];
         D.f[BNE ] = &DD[BNE *size_Mat];
         D.f[BSW ] = &DD[BSW *size_Mat];
         D.f[BSE ] = &DD[BSE *size_Mat];
         D.f[BNW ] = &DD[BNW *size_Mat];
      } 
      else
      {
         D.f[W   ] = &DD[E   *size_Mat];
         D.f[E   ] = &DD[W   *size_Mat];
         D.f[S   ] = &DD[N   *size_Mat];
         D.f[N   ] = &DD[S   *size_Mat];
         D.f[B   ] = &DD[T   *size_Mat];
         D.f[T   ] = &DD[B   *size_Mat];
         D.f[SW  ] = &DD[NE  *size_Mat];
         D.f[NE  ] = &DD[SW  *size_Mat];
         D.f[NW  ] = &DD[SE  *size_Mat];
         D.f[SE  ] = &DD[NW  *size_Mat];
         D.f[BW  ] = &DD[TE  *size_Mat];
         D.f[TE  ] = &DD[BW  *size_Mat];
         D.f[TW  ] = &DD[BE  *size_Mat];
         D.f[BE  ] = &DD[TW  *size_Mat];
         D.f[BS  ] = &DD[TN  *size_Mat];
         D.f[TN  ] = &DD[BS  *size_Mat];
         D.f[TS  ] = &DD[BN  *size_Mat];
         D.f[BN  ] = &DD[TS  *size_Mat];
         D.f[REST] = &DD[REST*size_Mat];
         D.f[TNE ] = &DD[BSW *size_Mat];
         D.f[TSW ] = &DD[BNE *size_Mat];
         D.f[TSE ] = &DD[BNW *size_Mat];
         D.f[TNW ] = &DD[BSE *size_Mat];
         D.f[BNE ] = &DD[TSW *size_Mat];
         D.f[BSW ] = &DD[TNE *size_Mat];
         D.f[BSE ] = &DD[TNW *size_Mat];
         D.f[BNW ] = &DD[TSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[REST])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real om_turb = om1 / (c1o1 + c3o1*om1*max(c0o1, turbViscosity[k_Q[k]]));
     
     real fac = c1o1;//c99o100;
	  real VeloX = fac*vx1;
	  real VeloY = fac*vx2;
	  real VeloZ = fac*vx3;
	  bool x = false;
	  bool y = false;
	  bool z = false;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[W])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cu_sq); 
         //(D.f[W])[kw]=(one-q)/(one+q)*(f_E-feq*om1)/(one-om1)+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);
         //(D.f[W])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = c0o1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[E])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cu_sq); 
         //(D.f[E])[ke]=(one-q)/(one+q)*(f_W-feq*om_turb)/(one-om_turb)+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);
         //(D.f[E])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[S])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cu_sq); 
         //(D.f[S])[ks]=(one-q)/(one+q)*(f_N-feq*om_turb)/(one-om_turb)+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);
         //(D.f[S])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
		 VeloY = c0o1;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[N])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cu_sq); 
         //(D.f[N])[kn]=(one-q)/(one+q)*(f_S-feq*om_turb)/(one-om_turb)+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);
         //(D.f[N])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[B])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cu_sq); 
         //(D.f[B])[kb]=(one-q)/(one+q)*(f_T-feq*om_turb)/(one-om_turb)+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);
         //(D.f[B])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = c0o1;
		 z = true;
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[T])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
         //feq=c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cu_sq); 
         //(D.f[T])[kt]=(one-q)/(one+q)*(f_B-feq*om_turb)/(one-om_turb)+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);
         //(D.f[T])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[SW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         //(D.f[SW])[ksw]=(one-q)/(one+q)*(f_NE-feq*om_turb)/(one-om_turb)+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);
         //(D.f[SW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[NE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         //(D.f[NE])[kne]=(one-q)/(one+q)*(f_SW-feq*om_turb)/(one-om_turb)+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);
         //(D.f[NE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[NW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         //(D.f[NW])[knw]=(one-q)/(one+q)*(f_SE-feq*om_turb)/(one-om_turb)+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);
         //(D.f[NW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[SE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         //(D.f[SE])[kse]=(one-q)/(one+q)*(f_NW-feq*om_turb)/(one-om_turb)+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);
         //(D.f[SE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
      //  if (k==10000) printf("AFTER x: %u \t  y: %u \t z: %u \n  VeloX: %f \t VeloY: %f \t VeloZ: %f \n\n", x,y,z, VeloX,VeloY,VeloZ);
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         //(D.f[BW])[kbw]=(one-q)/(one+q)*(f_TE-feq*om_turb)/(one-om_turb)+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);
         //(D.f[BW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         //(D.f[TE])[kte]=(one-q)/(one+q)*(f_BW-feq*om_turb)/(one-om_turb)+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);
         //(D.f[TE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         //(D.f[TW])[ktw]=(one-q)/(one+q)*(f_BE-feq*om_turb)/(one-om_turb)+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);
         //(D.f[TW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         //(D.f[BE])[kbe]=(one-q)/(one+q)*(f_TW-feq*om_turb)/(one-om_turb)+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);
         //(D.f[BE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         //(D.f[BS])[kbs]=(one-q)/(one+q)*(f_TN-feq*om_turb)/(one-om_turb)+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);
         //(D.f[BS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         //(D.f[TN])[ktn]=(one-q)/(one+q)*(f_BS-feq*om_turb)/(one-om_turb)+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);
         //(D.f[TN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         //(D.f[TS])[kts]=(one-q)/(one+q)*(f_BN-feq*om_turb)/(one-om_turb)+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);
         //(D.f[TS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
         //feq=c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         //(D.f[BN])[kbn]=(one-q)/(one+q)*(f_TS-feq*om_turb)/(one-om_turb)+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);
         //(D.f[BN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         //(D.f[BSW])[kbsw]=(one-q)/(one+q)*(f_TNE-feq*om_turb)/(one-om_turb)+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[BSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         //(D.f[TNE])[ktne]=(one-q)/(one+q)*(f_BSW-feq*om_turb)/(one-om_turb)+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[TNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         //(D.f[TSW])[ktsw]=(one-q)/(one+q)*(f_BNE-feq*om_turb)/(one-om_turb)+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[TSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         //(D.f[BNE])[kbne]=(one-q)/(one+q)*(f_TSW-feq*om_turb)/(one-om_turb)+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[BNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         //(D.f[BNW])[kbnw]=(one-q)/(one+q)*(f_TSE-feq*om_turb)/(one-om_turb)+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[BNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         //(D.f[TSE])[ktse]=(one-q)/(one+q)*(f_BNW-feq*om_turb)/(one-om_turb)+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[TSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         //(D.f[TNW])[ktnw]=(one-q)/(one+q)*(f_BSE-feq*om_turb)/(one-om_turb)+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[TNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = c0o1;
		 if (y == true) VeloY = c0o1;
		 if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //feq=c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         //(D.f[BSE])[kbse]=(one-q)/(one+q)*(f_TNW-feq*om_turb)/(one-om_turb)+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[BSE])[kbse]=zero;
      }
   }
}






































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipGeomDeviceComp27(real* DD, 
												 int* k_Q, 
												 real* QQ,
												 unsigned int  numberOfBCnodes,
												 real om1, 
												 real* NormalX,
												 real* NormalY,
												 real* NormalZ,
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[E   ] = &DD[E   *size_Mat];
      D.f[W   ] = &DD[W   *size_Mat];
      D.f[N   ] = &DD[N   *size_Mat];
      D.f[S   ] = &DD[S   *size_Mat];
      D.f[T   ] = &DD[T   *size_Mat];
      D.f[B   ] = &DD[B   *size_Mat];
      D.f[NE  ] = &DD[NE  *size_Mat];
      D.f[SW  ] = &DD[SW  *size_Mat];
      D.f[SE  ] = &DD[SE  *size_Mat];
      D.f[NW  ] = &DD[NW  *size_Mat];
      D.f[TE  ] = &DD[TE  *size_Mat];
      D.f[BW  ] = &DD[BW  *size_Mat];
      D.f[BE  ] = &DD[BE  *size_Mat];
      D.f[TW  ] = &DD[TW  *size_Mat];
      D.f[TN  ] = &DD[TN  *size_Mat];
      D.f[BS  ] = &DD[BS  *size_Mat];
      D.f[BN  ] = &DD[BN  *size_Mat];
      D.f[TS  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[TNE *size_Mat];
      D.f[TSW ] = &DD[TSW *size_Mat];
      D.f[TSE ] = &DD[TSE *size_Mat];
      D.f[TNW ] = &DD[TNW *size_Mat];
      D.f[BNE ] = &DD[BNE *size_Mat];
      D.f[BSW ] = &DD[BSW *size_Mat];
      D.f[BSE ] = &DD[BSE *size_Mat];
      D.f[BNW ] = &DD[BNW *size_Mat];
   } 
   else
   {
      D.f[W   ] = &DD[E   *size_Mat];
      D.f[E   ] = &DD[W   *size_Mat];
      D.f[S   ] = &DD[N   *size_Mat];
      D.f[N   ] = &DD[S   *size_Mat];
      D.f[B   ] = &DD[T   *size_Mat];
      D.f[T   ] = &DD[B   *size_Mat];
      D.f[SW  ] = &DD[NE  *size_Mat];
      D.f[NE  ] = &DD[SW  *size_Mat];
      D.f[NW  ] = &DD[SE  *size_Mat];
      D.f[SE  ] = &DD[NW  *size_Mat];
      D.f[BW  ] = &DD[TE  *size_Mat];
      D.f[TE  ] = &DD[BW  *size_Mat];
      D.f[TW  ] = &DD[BE  *size_Mat];
      D.f[BE  ] = &DD[TW  *size_Mat];
      D.f[BS  ] = &DD[TN  *size_Mat];
      D.f[TN  ] = &DD[BS  *size_Mat];
      D.f[TS  ] = &DD[BN  *size_Mat];
      D.f[BN  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[BSW *size_Mat];
      D.f[TSW ] = &DD[BNE *size_Mat];
      D.f[TSE ] = &DD[BNW *size_Mat];
      D.f[TNW ] = &DD[BSE *size_Mat];
      D.f[BNE ] = &DD[TSW *size_Mat];
      D.f[BSW ] = &DD[TNE *size_Mat];
      D.f[BSE ] = &DD[TNW *size_Mat];
      D.f[BNW ] = &DD[TSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[E   * numberOfBCnodes];
      q_dirW   = &QQ[W   * numberOfBCnodes];
      q_dirN   = &QQ[N   * numberOfBCnodes];
      q_dirS   = &QQ[S   * numberOfBCnodes];
      q_dirT   = &QQ[T   * numberOfBCnodes];
      q_dirB   = &QQ[B   * numberOfBCnodes];
      q_dirNE  = &QQ[NE  * numberOfBCnodes];
      q_dirSW  = &QQ[SW  * numberOfBCnodes];
      q_dirSE  = &QQ[SE  * numberOfBCnodes];
      q_dirNW  = &QQ[NW  * numberOfBCnodes];
      q_dirTE  = &QQ[TE  * numberOfBCnodes];
      q_dirBW  = &QQ[BW  * numberOfBCnodes];
      q_dirBE  = &QQ[BE  * numberOfBCnodes];
      q_dirTW  = &QQ[TW  * numberOfBCnodes];
      q_dirTN  = &QQ[TN  * numberOfBCnodes];
      q_dirBS  = &QQ[BS  * numberOfBCnodes];
      q_dirBN  = &QQ[BN  * numberOfBCnodes];
      q_dirTS  = &QQ[TS  * numberOfBCnodes];
      q_dirTNE = &QQ[TNE * numberOfBCnodes];
      q_dirTSW = &QQ[TSW * numberOfBCnodes];
      q_dirTSE = &QQ[TSE * numberOfBCnodes];
      q_dirTNW = &QQ[TNW * numberOfBCnodes];
      q_dirBNE = &QQ[BNE * numberOfBCnodes];
      q_dirBSW = &QQ[BSW * numberOfBCnodes];
      q_dirBSE = &QQ[BSE * numberOfBCnodes];
      q_dirBNW = &QQ[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[E   * numberOfBCnodes];
      nx_dirW   = &NormalX[W   * numberOfBCnodes];
      nx_dirN   = &NormalX[N   * numberOfBCnodes];
      nx_dirS   = &NormalX[S   * numberOfBCnodes];
      nx_dirT   = &NormalX[T   * numberOfBCnodes];
      nx_dirB   = &NormalX[B   * numberOfBCnodes];
      nx_dirNE  = &NormalX[NE  * numberOfBCnodes];
      nx_dirSW  = &NormalX[SW  * numberOfBCnodes];
      nx_dirSE  = &NormalX[SE  * numberOfBCnodes];
      nx_dirNW  = &NormalX[NW  * numberOfBCnodes];
      nx_dirTE  = &NormalX[TE  * numberOfBCnodes];
      nx_dirBW  = &NormalX[BW  * numberOfBCnodes];
      nx_dirBE  = &NormalX[BE  * numberOfBCnodes];
      nx_dirTW  = &NormalX[TW  * numberOfBCnodes];
      nx_dirTN  = &NormalX[TN  * numberOfBCnodes];
      nx_dirBS  = &NormalX[BS  * numberOfBCnodes];
      nx_dirBN  = &NormalX[BN  * numberOfBCnodes];
      nx_dirTS  = &NormalX[TS  * numberOfBCnodes];
      nx_dirTNE = &NormalX[TNE * numberOfBCnodes];
      nx_dirTSW = &NormalX[TSW * numberOfBCnodes];
      nx_dirTSE = &NormalX[TSE * numberOfBCnodes];
      nx_dirTNW = &NormalX[TNW * numberOfBCnodes];
      nx_dirBNE = &NormalX[BNE * numberOfBCnodes];
      nx_dirBSW = &NormalX[BSW * numberOfBCnodes];
      nx_dirBSE = &NormalX[BSE * numberOfBCnodes];
      nx_dirBNW = &NormalX[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[E   * numberOfBCnodes];
      ny_dirW   = &NormalY[W   * numberOfBCnodes];
      ny_dirN   = &NormalY[N   * numberOfBCnodes];
      ny_dirS   = &NormalY[S   * numberOfBCnodes];
      ny_dirT   = &NormalY[T   * numberOfBCnodes];
      ny_dirB   = &NormalY[B   * numberOfBCnodes];
      ny_dirNE  = &NormalY[NE  * numberOfBCnodes];
      ny_dirSW  = &NormalY[SW  * numberOfBCnodes];
      ny_dirSE  = &NormalY[SE  * numberOfBCnodes];
      ny_dirNW  = &NormalY[NW  * numberOfBCnodes];
      ny_dirTE  = &NormalY[TE  * numberOfBCnodes];
      ny_dirBW  = &NormalY[BW  * numberOfBCnodes];
      ny_dirBE  = &NormalY[BE  * numberOfBCnodes];
      ny_dirTW  = &NormalY[TW  * numberOfBCnodes];
      ny_dirTN  = &NormalY[TN  * numberOfBCnodes];
      ny_dirBS  = &NormalY[BS  * numberOfBCnodes];
      ny_dirBN  = &NormalY[BN  * numberOfBCnodes];
      ny_dirTS  = &NormalY[TS  * numberOfBCnodes];
      ny_dirTNE = &NormalY[TNE * numberOfBCnodes];
      ny_dirTSW = &NormalY[TSW * numberOfBCnodes];
      ny_dirTSE = &NormalY[TSE * numberOfBCnodes];
      ny_dirTNW = &NormalY[TNW * numberOfBCnodes];
      ny_dirBNE = &NormalY[BNE * numberOfBCnodes];
      ny_dirBSW = &NormalY[BSW * numberOfBCnodes];
      ny_dirBSE = &NormalY[BSE * numberOfBCnodes];
      ny_dirBNW = &NormalY[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[E   * numberOfBCnodes];
      nz_dirW   = &NormalZ[W   * numberOfBCnodes];
      nz_dirN   = &NormalZ[N   * numberOfBCnodes];
      nz_dirS   = &NormalZ[S   * numberOfBCnodes];
      nz_dirT   = &NormalZ[T   * numberOfBCnodes];
      nz_dirB   = &NormalZ[B   * numberOfBCnodes];
      nz_dirNE  = &NormalZ[NE  * numberOfBCnodes];
      nz_dirSW  = &NormalZ[SW  * numberOfBCnodes];
      nz_dirSE  = &NormalZ[SE  * numberOfBCnodes];
      nz_dirNW  = &NormalZ[NW  * numberOfBCnodes];
      nz_dirTE  = &NormalZ[TE  * numberOfBCnodes];
      nz_dirBW  = &NormalZ[BW  * numberOfBCnodes];
      nz_dirBE  = &NormalZ[BE  * numberOfBCnodes];
      nz_dirTW  = &NormalZ[TW  * numberOfBCnodes];
      nz_dirTN  = &NormalZ[TN  * numberOfBCnodes];
      nz_dirBS  = &NormalZ[BS  * numberOfBCnodes];
      nz_dirBN  = &NormalZ[BN  * numberOfBCnodes];
      nz_dirTS  = &NormalZ[TS  * numberOfBCnodes];
      nz_dirTNE = &NormalZ[TNE * numberOfBCnodes];
      nz_dirTSW = &NormalZ[TSW * numberOfBCnodes];
      nz_dirTSE = &NormalZ[TSE * numberOfBCnodes];
      nz_dirTNW = &NormalZ[TNW * numberOfBCnodes];
      nz_dirBNE = &NormalZ[BNE * numberOfBCnodes];
      nz_dirBSW = &NormalZ[BSW * numberOfBCnodes];
      nz_dirBSE = &NormalZ[BSE * numberOfBCnodes];
      nz_dirBNW = &NormalZ[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[E   ])[ke   ];
      real f_E    = (D.f[W   ])[kw   ];
      real f_S    = (D.f[N   ])[kn   ];
      real f_N    = (D.f[S   ])[ks   ];
      real f_B    = (D.f[T   ])[kt   ];
      real f_T    = (D.f[B   ])[kb   ];
      real f_SW   = (D.f[NE  ])[kne  ];
      real f_NE   = (D.f[SW  ])[ksw  ];
      real f_NW   = (D.f[SE  ])[kse  ];
      real f_SE   = (D.f[NW  ])[knw  ];
      real f_BW   = (D.f[TE  ])[kte  ];
      real f_TE   = (D.f[BW  ])[kbw  ];
      real f_TW   = (D.f[BE  ])[kbe  ];
      real f_BE   = (D.f[TW  ])[ktw  ];
      real f_BS   = (D.f[TN  ])[ktn  ];
      real f_TN   = (D.f[BS  ])[kbs  ];
      real f_TS   = (D.f[BN  ])[kbn  ];
      real f_BN   = (D.f[TS  ])[kts  ];
      real f_BSW  = (D.f[TNE ])[ktne ];
      real f_BNE  = (D.f[TSW ])[ktsw ];
      real f_BNW  = (D.f[TSE ])[ktse ];
      real f_BSE  = (D.f[TNW ])[ktnw ];
      real f_TSW  = (D.f[BNE ])[kbne ];
      real f_TNE  = (D.f[BSW ])[kbsw ];
      real f_TNW  = (D.f[BSE ])[kbse ];
      real f_TSE  = (D.f[BNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[REST])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[E   ] = &DD[E   *size_Mat];
         D.f[W   ] = &DD[W   *size_Mat];
         D.f[N   ] = &DD[N   *size_Mat];
         D.f[S   ] = &DD[S   *size_Mat];
         D.f[T   ] = &DD[T   *size_Mat];
         D.f[B   ] = &DD[B   *size_Mat];
         D.f[NE  ] = &DD[NE  *size_Mat];
         D.f[SW  ] = &DD[SW  *size_Mat];
         D.f[SE  ] = &DD[SE  *size_Mat];
         D.f[NW  ] = &DD[NW  *size_Mat];
         D.f[TE  ] = &DD[TE  *size_Mat];
         D.f[BW  ] = &DD[BW  *size_Mat];
         D.f[BE  ] = &DD[BE  *size_Mat];
         D.f[TW  ] = &DD[TW  *size_Mat];
         D.f[TN  ] = &DD[TN  *size_Mat];
         D.f[BS  ] = &DD[BS  *size_Mat];
         D.f[BN  ] = &DD[BN  *size_Mat];
         D.f[TS  ] = &DD[TS  *size_Mat];
         D.f[REST] = &DD[REST*size_Mat];
         D.f[TNE ] = &DD[TNE *size_Mat];
         D.f[TSW ] = &DD[TSW *size_Mat];
         D.f[TSE ] = &DD[TSE *size_Mat];
         D.f[TNW ] = &DD[TNW *size_Mat];
         D.f[BNE ] = &DD[BNE *size_Mat];
         D.f[BSW ] = &DD[BSW *size_Mat];
         D.f[BSE ] = &DD[BSE *size_Mat];
         D.f[BNW ] = &DD[BNW *size_Mat];
      } 
      else
      {
         D.f[W   ] = &DD[E   *size_Mat];
         D.f[E   ] = &DD[W   *size_Mat];
         D.f[S   ] = &DD[N   *size_Mat];
         D.f[N   ] = &DD[S   *size_Mat];
         D.f[B   ] = &DD[T   *size_Mat];
         D.f[T   ] = &DD[B   *size_Mat];
         D.f[SW  ] = &DD[NE  *size_Mat];
         D.f[NE  ] = &DD[SW  *size_Mat];
         D.f[NW  ] = &DD[SE  *size_Mat];
         D.f[SE  ] = &DD[NW  *size_Mat];
         D.f[BW  ] = &DD[TE  *size_Mat];
         D.f[TE  ] = &DD[BW  *size_Mat];
         D.f[TW  ] = &DD[BE  *size_Mat];
         D.f[BE  ] = &DD[TW  *size_Mat];
         D.f[BS  ] = &DD[TN  *size_Mat];
         D.f[TN  ] = &DD[BS  *size_Mat];
         D.f[TS  ] = &DD[BN  *size_Mat];
         D.f[BN  ] = &DD[TS  *size_Mat];
         D.f[REST] = &DD[REST*size_Mat];
         D.f[TNE ] = &DD[BSW *size_Mat];
         D.f[TSW ] = &DD[BNE *size_Mat];
         D.f[TSE ] = &DD[BNW *size_Mat];
         D.f[TNW ] = &DD[BSE *size_Mat];
         D.f[BNE ] = &DD[TSW *size_Mat];
         D.f[BSW ] = &DD[TNE *size_Mat];
         D.f[BSE ] = &DD[TNW *size_Mat];
         D.f[BNW ] = &DD[TSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real VeloX = vx1;
	  real VeloY = vx2;
	  real VeloZ = vx3;
	  real fac = c0o1;//0.5;
 	  real phi = c0o1;
	  //real alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real kxyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho) - ((vx1*vx2)));
      real kyzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho) - ((vx2*vx3)));
      real kxzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho) - ((vx1*vx3)));

	  real kxxFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx1*vx1)));
	  real kyyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx2*vx2)));
	  real kzzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx3*vx3)));

	  real magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

	  //fac = fac * magS / (c1o3 * (one / om1 - c1o2));
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real *facAst = &QQ[REST * numberOfBCnodes];

	  //fac = fac * alpha + facAst[k] * (one - alpha);
	  //facAst[k] = fac;
	  //(&QQ[REST * numberOfBCnodes])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////real uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real phi = expf(magS/0.01f) - one;
	  //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real C = five;
	  //real kappa = 0.41f;
	  //real phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
	  //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
	  //real qMed = sum/count;
	  //real phi = fac / (qMed + fac);
	  //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real testQ = c2o1;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirE[k]) + fac);
		 VeloX *= phi;
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[W])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirW[k]) + fac);
		 VeloX *= phi;
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[E])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirN[k]) + fac);
		 VeloY *= phi;
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[S])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirS[k]) + fac);
		 VeloY *= phi;
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[N])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nz_dirT[k]) + fac);
		 VeloZ *= phi;
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[B])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nz_dirB[k]) + fac);
		 VeloZ *= phi;
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[T])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
		 VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirNE[k] + ny_dirNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[SW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
		 VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirSW[k] - ny_dirSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[NE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
		 VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirSE[k] - ny_dirSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[NW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
		 VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirNW[k] + ny_dirNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[SE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTE[k] + nz_dirTE[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBW[k] - nz_dirBW[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBE[k] - nz_dirBE[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTW[k] + nz_dirTW[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
		 VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirTN[k] + nz_dirTN[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
		 VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirBS[k] - nz_dirBS[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
		 VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirBN[k] - nz_dirBN[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
		 VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirTS[k] + nz_dirTS[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
		 VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
		 VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
		 VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
		 VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
		 VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
		 VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
		 VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = (phi > one) ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
		 VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipNormDeviceComp27(real* DD, 
												 int* k_Q, 
												 real* QQ,
												 unsigned int  numberOfBCnodes,
												 real om1, 
												 real* NormalX,
												 real* NormalY,
												 real* NormalZ,
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[E   ] = &DD[E   *size_Mat];
      D.f[W   ] = &DD[W   *size_Mat];
      D.f[N   ] = &DD[N   *size_Mat];
      D.f[S   ] = &DD[S   *size_Mat];
      D.f[T   ] = &DD[T   *size_Mat];
      D.f[B   ] = &DD[B   *size_Mat];
      D.f[NE  ] = &DD[NE  *size_Mat];
      D.f[SW  ] = &DD[SW  *size_Mat];
      D.f[SE  ] = &DD[SE  *size_Mat];
      D.f[NW  ] = &DD[NW  *size_Mat];
      D.f[TE  ] = &DD[TE  *size_Mat];
      D.f[BW  ] = &DD[BW  *size_Mat];
      D.f[BE  ] = &DD[BE  *size_Mat];
      D.f[TW  ] = &DD[TW  *size_Mat];
      D.f[TN  ] = &DD[TN  *size_Mat];
      D.f[BS  ] = &DD[BS  *size_Mat];
      D.f[BN  ] = &DD[BN  *size_Mat];
      D.f[TS  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[TNE *size_Mat];
      D.f[TSW ] = &DD[TSW *size_Mat];
      D.f[TSE ] = &DD[TSE *size_Mat];
      D.f[TNW ] = &DD[TNW *size_Mat];
      D.f[BNE ] = &DD[BNE *size_Mat];
      D.f[BSW ] = &DD[BSW *size_Mat];
      D.f[BSE ] = &DD[BSE *size_Mat];
      D.f[BNW ] = &DD[BNW *size_Mat];
   } 
   else
   {
      D.f[W   ] = &DD[E   *size_Mat];
      D.f[E   ] = &DD[W   *size_Mat];
      D.f[S   ] = &DD[N   *size_Mat];
      D.f[N   ] = &DD[S   *size_Mat];
      D.f[B   ] = &DD[T   *size_Mat];
      D.f[T   ] = &DD[B   *size_Mat];
      D.f[SW  ] = &DD[NE  *size_Mat];
      D.f[NE  ] = &DD[SW  *size_Mat];
      D.f[NW  ] = &DD[SE  *size_Mat];
      D.f[SE  ] = &DD[NW  *size_Mat];
      D.f[BW  ] = &DD[TE  *size_Mat];
      D.f[TE  ] = &DD[BW  *size_Mat];
      D.f[TW  ] = &DD[BE  *size_Mat];
      D.f[BE  ] = &DD[TW  *size_Mat];
      D.f[BS  ] = &DD[TN  *size_Mat];
      D.f[TN  ] = &DD[BS  *size_Mat];
      D.f[TS  ] = &DD[BN  *size_Mat];
      D.f[BN  ] = &DD[TS  *size_Mat];
      D.f[REST] = &DD[REST*size_Mat];
      D.f[TNE ] = &DD[BSW *size_Mat];
      D.f[TSW ] = &DD[BNE *size_Mat];
      D.f[TSE ] = &DD[BNW *size_Mat];
      D.f[TNW ] = &DD[BSE *size_Mat];
      D.f[BNE ] = &DD[TSW *size_Mat];
      D.f[BSW ] = &DD[TNE *size_Mat];
      D.f[BSE ] = &DD[TNW *size_Mat];
      D.f[BNW ] = &DD[TSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[E   * numberOfBCnodes];
      q_dirW   = &QQ[W   * numberOfBCnodes];
      q_dirN   = &QQ[N   * numberOfBCnodes];
      q_dirS   = &QQ[S   * numberOfBCnodes];
      q_dirT   = &QQ[T   * numberOfBCnodes];
      q_dirB   = &QQ[B   * numberOfBCnodes];
      q_dirNE  = &QQ[NE  * numberOfBCnodes];
      q_dirSW  = &QQ[SW  * numberOfBCnodes];
      q_dirSE  = &QQ[SE  * numberOfBCnodes];
      q_dirNW  = &QQ[NW  * numberOfBCnodes];
      q_dirTE  = &QQ[TE  * numberOfBCnodes];
      q_dirBW  = &QQ[BW  * numberOfBCnodes];
      q_dirBE  = &QQ[BE  * numberOfBCnodes];
      q_dirTW  = &QQ[TW  * numberOfBCnodes];
      q_dirTN  = &QQ[TN  * numberOfBCnodes];
      q_dirBS  = &QQ[BS  * numberOfBCnodes];
      q_dirBN  = &QQ[BN  * numberOfBCnodes];
      q_dirTS  = &QQ[TS  * numberOfBCnodes];
      q_dirTNE = &QQ[TNE * numberOfBCnodes];
      q_dirTSW = &QQ[TSW * numberOfBCnodes];
      q_dirTSE = &QQ[TSE * numberOfBCnodes];
      q_dirTNW = &QQ[TNW * numberOfBCnodes];
      q_dirBNE = &QQ[BNE * numberOfBCnodes];
      q_dirBSW = &QQ[BSW * numberOfBCnodes];
      q_dirBSE = &QQ[BSE * numberOfBCnodes];
      q_dirBNW = &QQ[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[E   * numberOfBCnodes];
      nx_dirW   = &NormalX[W   * numberOfBCnodes];
      nx_dirN   = &NormalX[N   * numberOfBCnodes];
      nx_dirS   = &NormalX[S   * numberOfBCnodes];
      nx_dirT   = &NormalX[T   * numberOfBCnodes];
      nx_dirB   = &NormalX[B   * numberOfBCnodes];
      nx_dirNE  = &NormalX[NE  * numberOfBCnodes];
      nx_dirSW  = &NormalX[SW  * numberOfBCnodes];
      nx_dirSE  = &NormalX[SE  * numberOfBCnodes];
      nx_dirNW  = &NormalX[NW  * numberOfBCnodes];
      nx_dirTE  = &NormalX[TE  * numberOfBCnodes];
      nx_dirBW  = &NormalX[BW  * numberOfBCnodes];
      nx_dirBE  = &NormalX[BE  * numberOfBCnodes];
      nx_dirTW  = &NormalX[TW  * numberOfBCnodes];
      nx_dirTN  = &NormalX[TN  * numberOfBCnodes];
      nx_dirBS  = &NormalX[BS  * numberOfBCnodes];
      nx_dirBN  = &NormalX[BN  * numberOfBCnodes];
      nx_dirTS  = &NormalX[TS  * numberOfBCnodes];
      nx_dirTNE = &NormalX[TNE * numberOfBCnodes];
      nx_dirTSW = &NormalX[TSW * numberOfBCnodes];
      nx_dirTSE = &NormalX[TSE * numberOfBCnodes];
      nx_dirTNW = &NormalX[TNW * numberOfBCnodes];
      nx_dirBNE = &NormalX[BNE * numberOfBCnodes];
      nx_dirBSW = &NormalX[BSW * numberOfBCnodes];
      nx_dirBSE = &NormalX[BSE * numberOfBCnodes];
      nx_dirBNW = &NormalX[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[E   * numberOfBCnodes];
      ny_dirW   = &NormalY[W   * numberOfBCnodes];
      ny_dirN   = &NormalY[N   * numberOfBCnodes];
      ny_dirS   = &NormalY[S   * numberOfBCnodes];
      ny_dirT   = &NormalY[T   * numberOfBCnodes];
      ny_dirB   = &NormalY[B   * numberOfBCnodes];
      ny_dirNE  = &NormalY[NE  * numberOfBCnodes];
      ny_dirSW  = &NormalY[SW  * numberOfBCnodes];
      ny_dirSE  = &NormalY[SE  * numberOfBCnodes];
      ny_dirNW  = &NormalY[NW  * numberOfBCnodes];
      ny_dirTE  = &NormalY[TE  * numberOfBCnodes];
      ny_dirBW  = &NormalY[BW  * numberOfBCnodes];
      ny_dirBE  = &NormalY[BE  * numberOfBCnodes];
      ny_dirTW  = &NormalY[TW  * numberOfBCnodes];
      ny_dirTN  = &NormalY[TN  * numberOfBCnodes];
      ny_dirBS  = &NormalY[BS  * numberOfBCnodes];
      ny_dirBN  = &NormalY[BN  * numberOfBCnodes];
      ny_dirTS  = &NormalY[TS  * numberOfBCnodes];
      ny_dirTNE = &NormalY[TNE * numberOfBCnodes];
      ny_dirTSW = &NormalY[TSW * numberOfBCnodes];
      ny_dirTSE = &NormalY[TSE * numberOfBCnodes];
      ny_dirTNW = &NormalY[TNW * numberOfBCnodes];
      ny_dirBNE = &NormalY[BNE * numberOfBCnodes];
      ny_dirBSW = &NormalY[BSW * numberOfBCnodes];
      ny_dirBSE = &NormalY[BSE * numberOfBCnodes];
      ny_dirBNW = &NormalY[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[E   * numberOfBCnodes];
      nz_dirW   = &NormalZ[W   * numberOfBCnodes];
      nz_dirN   = &NormalZ[N   * numberOfBCnodes];
      nz_dirS   = &NormalZ[S   * numberOfBCnodes];
      nz_dirT   = &NormalZ[T   * numberOfBCnodes];
      nz_dirB   = &NormalZ[B   * numberOfBCnodes];
      nz_dirNE  = &NormalZ[NE  * numberOfBCnodes];
      nz_dirSW  = &NormalZ[SW  * numberOfBCnodes];
      nz_dirSE  = &NormalZ[SE  * numberOfBCnodes];
      nz_dirNW  = &NormalZ[NW  * numberOfBCnodes];
      nz_dirTE  = &NormalZ[TE  * numberOfBCnodes];
      nz_dirBW  = &NormalZ[BW  * numberOfBCnodes];
      nz_dirBE  = &NormalZ[BE  * numberOfBCnodes];
      nz_dirTW  = &NormalZ[TW  * numberOfBCnodes];
      nz_dirTN  = &NormalZ[TN  * numberOfBCnodes];
      nz_dirBS  = &NormalZ[BS  * numberOfBCnodes];
      nz_dirBN  = &NormalZ[BN  * numberOfBCnodes];
      nz_dirTS  = &NormalZ[TS  * numberOfBCnodes];
      nz_dirTNE = &NormalZ[TNE * numberOfBCnodes];
      nz_dirTSW = &NormalZ[TSW * numberOfBCnodes];
      nz_dirTSE = &NormalZ[TSE * numberOfBCnodes];
      nz_dirTNW = &NormalZ[TNW * numberOfBCnodes];
      nz_dirBNE = &NormalZ[BNE * numberOfBCnodes];
      nz_dirBSW = &NormalZ[BSW * numberOfBCnodes];
      nz_dirBSE = &NormalZ[BSE * numberOfBCnodes];
      nz_dirBNW = &NormalZ[BNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[E   ])[ke   ];
      real f_E    = (D.f[W   ])[kw   ];
      real f_S    = (D.f[N   ])[kn   ];
      real f_N    = (D.f[S   ])[ks   ];
      real f_B    = (D.f[T   ])[kt   ];
      real f_T    = (D.f[B   ])[kb   ];
      real f_SW   = (D.f[NE  ])[kne  ];
      real f_NE   = (D.f[SW  ])[ksw  ];
      real f_NW   = (D.f[SE  ])[kse  ];
      real f_SE   = (D.f[NW  ])[knw  ];
      real f_BW   = (D.f[TE  ])[kte  ];
      real f_TE   = (D.f[BW  ])[kbw  ];
      real f_TW   = (D.f[BE  ])[kbe  ];
      real f_BE   = (D.f[TW  ])[ktw  ];
      real f_BS   = (D.f[TN  ])[ktn  ];
      real f_TN   = (D.f[BS  ])[kbs  ];
      real f_TS   = (D.f[BN  ])[kbn  ];
      real f_BN   = (D.f[TS  ])[kts  ];
      real f_BSW  = (D.f[TNE ])[ktne ];
      real f_BNE  = (D.f[TSW ])[ktsw ];
      real f_BNW  = (D.f[TSE ])[ktse ];
      real f_BSE  = (D.f[TNW ])[ktnw ];
      real f_TSW  = (D.f[BNE ])[kbne ];
      real f_TNE  = (D.f[BSW ])[kbsw ];
      real f_TNW  = (D.f[BSE ])[kbse ];
      real f_TSE  = (D.f[BNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[REST])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[E   ] = &DD[E   *size_Mat];
         D.f[W   ] = &DD[W   *size_Mat];
         D.f[N   ] = &DD[N   *size_Mat];
         D.f[S   ] = &DD[S   *size_Mat];
         D.f[T   ] = &DD[T   *size_Mat];
         D.f[B   ] = &DD[B   *size_Mat];
         D.f[NE  ] = &DD[NE  *size_Mat];
         D.f[SW  ] = &DD[SW  *size_Mat];
         D.f[SE  ] = &DD[SE  *size_Mat];
         D.f[NW  ] = &DD[NW  *size_Mat];
         D.f[TE  ] = &DD[TE  *size_Mat];
         D.f[BW  ] = &DD[BW  *size_Mat];
         D.f[BE  ] = &DD[BE  *size_Mat];
         D.f[TW  ] = &DD[TW  *size_Mat];
         D.f[TN  ] = &DD[TN  *size_Mat];
         D.f[BS  ] = &DD[BS  *size_Mat];
         D.f[BN  ] = &DD[BN  *size_Mat];
         D.f[TS  ] = &DD[TS  *size_Mat];
         D.f[REST] = &DD[REST*size_Mat];
         D.f[TNE ] = &DD[TNE *size_Mat];
         D.f[TSW ] = &DD[TSW *size_Mat];
         D.f[TSE ] = &DD[TSE *size_Mat];
         D.f[TNW ] = &DD[TNW *size_Mat];
         D.f[BNE ] = &DD[BNE *size_Mat];
         D.f[BSW ] = &DD[BSW *size_Mat];
         D.f[BSE ] = &DD[BSE *size_Mat];
         D.f[BNW ] = &DD[BNW *size_Mat];
      } 
      else
      {
         D.f[W   ] = &DD[E   *size_Mat];
         D.f[E   ] = &DD[W   *size_Mat];
         D.f[S   ] = &DD[N   *size_Mat];
         D.f[N   ] = &DD[S   *size_Mat];
         D.f[B   ] = &DD[T   *size_Mat];
         D.f[T   ] = &DD[B   *size_Mat];
         D.f[SW  ] = &DD[NE  *size_Mat];
         D.f[NE  ] = &DD[SW  *size_Mat];
         D.f[NW  ] = &DD[SE  *size_Mat];
         D.f[SE  ] = &DD[NW  *size_Mat];
         D.f[BW  ] = &DD[TE  *size_Mat];
         D.f[TE  ] = &DD[BW  *size_Mat];
         D.f[TW  ] = &DD[BE  *size_Mat];
         D.f[BE  ] = &DD[TW  *size_Mat];
         D.f[BS  ] = &DD[TN  *size_Mat];
         D.f[TN  ] = &DD[BS  *size_Mat];
         D.f[TS  ] = &DD[BN  *size_Mat];
         D.f[BN  ] = &DD[TS  *size_Mat];
         D.f[REST] = &DD[REST*size_Mat];
         D.f[TNE ] = &DD[BSW *size_Mat];
         D.f[TSW ] = &DD[BNE *size_Mat];
         D.f[TSE ] = &DD[BNW *size_Mat];
         D.f[TNW ] = &DD[BSE *size_Mat];
         D.f[BNE ] = &DD[TSW *size_Mat];
         D.f[BSW ] = &DD[TNE *size_Mat];
         D.f[BSE ] = &DD[TNW *size_Mat];
         D.f[BNW ] = &DD[TSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real VeloX = vx1;
	  real VeloY = vx2;
	  real VeloZ = vx3;
	  real fac = c1o100;//0.5;
 	  //real phi = c0o1;
	  real alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real kxyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho) - ((vx1*vx2)));
      real kyzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho) - ((vx2*vx3)));
      real kxzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho) - ((vx1*vx3)));

	  real kxxFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx1*vx1)));
	  real kyyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx2*vx2)));
	  real kzzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx3*vx3)));

	  real magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

	  fac = fac * magS / (c1o3 * (c1o1 / om1 - c1o2));
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real *facAst = &QQ[REST * numberOfBCnodes];

	  fac = fac * alpha + facAst[k] * (c1o1 - alpha);
	  facAst[k] = fac;
	  //(&QQ[REST * numberOfBCnodes])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////real uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real phi = expf(magS/0.01f) - one;
	  //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real C = five;
	  //real kappa = 0.41f;
	  //real phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
	  //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //real sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
	  //real qMed = sum/count;
	  //real phi = fac / (qMed + fac);
	  //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real sliplength = 0.9f;//c1o2;
	  real qSlip = c0o1;
	  real un = c0o1;
	  real ut = c0o1;
	  real tangential = c0o1;
	  //real smallSingle = Op0000002;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
		 un = fabs((vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k]);
		 ut = fabs(VeloX);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[W])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
		 un = fabs(-(vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k]);
		 ut = fabs(-VeloX);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[E])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
		 un = fabs( (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k]);
		 ut = fabs( VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[S])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
		 un = fabs(-(vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k]);
		 ut = fabs(-VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[N])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
		 un = fabs( (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k]);
		 ut = fabs( VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nz_dirT[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[B])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
		 un = fabs(-(vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k]);
		 ut = fabs(-VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nz_dirB[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[T])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
		 VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
		 un = fabs( (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k] + (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k]);
		 ut = fabs( VeloX + VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirNE[k]+ny_dirNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[SW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
		 VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
		 un = fabs(-(vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k] - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k]);
		 ut = fabs(-VeloX - VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirSW[k]-ny_dirSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[NE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
		 VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
		 un = fabs( (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k] - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k]);
		 ut = fabs( VeloX - VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirSE[k]-ny_dirSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[NW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
		 VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
		 un = fabs(-(vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k] + (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k]);
		 ut = fabs(-VeloX + VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirNW[k]+ny_dirNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[SE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
		 un = fabs( (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k] + (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k]);
		 ut = fabs( VeloX + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTE[k]+nz_dirTE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
		 un = fabs(-(vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k] - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k]);
		 ut = fabs(-VeloX - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBW[k]-nz_dirBW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
		 un = fabs( (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k] - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k]);
		 ut = fabs( VeloX - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBE[k]-nz_dirBE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
		 un = fabs(-(vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k] + (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k]);
		 ut = fabs(-VeloX + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTW[k]+nz_dirTW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
		 VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
		 un = fabs( (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k] + (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k]);
		 ut = fabs( VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirTN[k]+nz_dirTN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
		 VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
		 un = fabs(-(vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k] - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k]);
		 ut = fabs(-VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirBS[k]-nz_dirBS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
		 VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
		 un = fabs( (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k] - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k]);
		 ut = fabs( VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirBN[k]-nz_dirBN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
		 VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
		 un = fabs(-(vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k] + (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k]);
		 ut = fabs(-VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirTS[k]+nz_dirTS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
		 VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
		 un = fabs( (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k] 
				   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k] 
				   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k]);
		 ut = fabs( VeloX + VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
		 VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
		 un = fabs(-(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k] 
				   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k] 
				   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k]);
		 ut = fabs(-VeloX - VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
		 VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
		 un = fabs( (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k] 
				   +(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k] 
				   -(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k]);
		 ut = fabs( VeloX + VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
		 VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
		 un = fabs(-(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k] 
				   -(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k] 
				   +(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k]);
		 ut = fabs(-VeloX - VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
		 VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
		 un = fabs(+(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k] 
				   -(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k] 
				   +(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k]);
		 ut = fabs(+VeloX - VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
		 VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
		 un = fabs(-(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k] 
				   +(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k] 
				   -(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k]);
		 ut = fabs(-VeloX + VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
		 VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
		 un = fabs( (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k] 
				   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k] 
				   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k]);
		 ut = fabs( VeloX - VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[TNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
		 VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
		 un = fabs(-(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k] 
				   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k] 
				   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k]);
		 ut = fabs(-VeloX + VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (smallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[BSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q) - c1o216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
