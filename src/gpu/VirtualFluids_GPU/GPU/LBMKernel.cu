//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ /
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
// includes, cuda
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "LBM/LB.h"
#include "cuda/CudaGrid.h"

// includes, kernels
#include "GPU/GPU_Kernels.cuh"

#include "Parameter/Parameter.h"
//////////////////////////////////////////////////////////////////////////
void KernelCas27( unsigned int grid_nx,
                             unsigned int grid_ny,
                             unsigned int grid_nz,
                             real s9,
                             unsigned int* bcMatD,
                             unsigned int* neighborX,
                             unsigned int* neighborY,
                             unsigned int* neighborZ,
                             real* DD,
                             int size_Mat,
                             bool EvenOrOdd)
{
   dim3 threads       ( grid_nx, 1, 1 );
   dim3 grid          ( grid_ny, grid_nz );   // Gitter fuer Kollision und Propagation

      LB_Kernel_Casc27<<< grid, threads >>>( s9,
                                             bcMatD,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             DD,
                                             size_Mat,
                                             EvenOrOdd);
     getLastCudaError("LB_Kernel_Casc27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelCasSP27( unsigned int numberOfThreads,
                               real s9,
                               unsigned int* bcMatD,
                               unsigned int* neighborX,
                               unsigned int* neighborY,
                               unsigned int* neighborZ,
                               real* DD,
                               int size_Mat,
                               bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_Casc_SP_27<<< grid.grid, grid.threads >>>(s9,
                                                bcMatD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                DD,
                                                size_Mat,
                                                EvenOrOdd);
      getLastCudaError("LB_Kernel_Casc_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelCasSPMS27( unsigned int numberOfThreads,
                                 real s9,
                                 unsigned int* bcMatD,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 real* DD,
                                 int size_Mat,
                                 bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_Casc_SP_MS_27<<< grid.grid, grid.threads >>>(s9,
                                                   bcMatD,
                                                   neighborX,
                                                   neighborY,
                                                   neighborZ,
                                                   DD,
                                                   size_Mat,
                                                   EvenOrOdd);
      getLastCudaError("LB_Kernel_Casc_SP_MS_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelCasSPMSOHM27( unsigned int numberOfThreads,
                                    real s9,
                                    unsigned int* bcMatD,
                                    unsigned int* neighborX,
                                    unsigned int* neighborY,
                                    unsigned int* neighborZ,
                                    real* DD,
                                    int size_Mat,
                                    bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_Casc_SP_MS_OHM_27<<< grid.grid, grid.threads >>>(  s9,
                                                         bcMatD,
                                                         neighborX,
                                                         neighborY,
                                                         neighborZ,
                                                         DD,
                                                         size_Mat,
                                                         EvenOrOdd);
      getLastCudaError("LB_Kernel_Casc_SP_MS_OHM_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelKumCompSRTSP27(
	unsigned int numberOfThreads,
	real omega,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* DDStart,
	int size_Mat,
	int level,
	real* forces,
	bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

   LB_Kernel_Kum_New_Comp_SRT_SP_27 <<< grid.grid, grid.threads >>>(
	   omega,
	   bcMatD,
	   neighborX,
	   neighborY,
	   neighborZ,
	   DDStart,
	   size_Mat,
	   level,
	   forces,
	   EvenOrOdd);
      getLastCudaError("LB_Kernel_Kum_New_Comp_SRT_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelKum1hSP27(    unsigned int numberOfThreads,
									real omega,
									real deltaPhi,
									real angularVelocity,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* coordX,
									real* coordY,
									real* coordZ,
									real* DDStart,
									int size_Mat,
									bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

		LB_Kernel_Kum_1h_SP_27<<< grid.grid, grid.threads >>>(omega,
													deltaPhi,
													angularVelocity,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													coordX,
													coordY,
													coordZ,
													DDStart,
													size_Mat,
													EvenOrOdd);
		getLastCudaError("LB_Kernel_Kum_New_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelCascadeSP27(  unsigned int numberOfThreads,
									real s9,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* DD,
									int size_Mat,
									bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

		LB_Kernel_Cascade_SP_27<<< grid.grid, grid.threads >>>(s9,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													size_Mat,
													EvenOrOdd);
		getLastCudaError("LB_Kernel_Cascade_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelKumNewSP27(   unsigned int numberOfThreads,
									real s9,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* DD,
									int size_Mat,
									bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


		LB_Kernel_Kum_New_SP_27<<< grid.grid, grid.threads >>>(s9,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													size_Mat,
													EvenOrOdd);
		getLastCudaError("LB_Kernel_Kum_New_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelKumNewCompSP27(unsigned int numberOfThreads,
									real s9,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* DD,
									int size_Mat,
									int size_Array,
									int level,
									real* forces,
									bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

		//LB_Kernel_Kum_New_Comp_SP_27<<< grid.grid, grid.threads >>>(	s9,
		//													bcMatD,
		//													neighborX,
		//													neighborY,
		//													neighborZ,
		//													DD,
		//													size_Mat,
		//													level,
		//													forces,
		//													EvenOrOdd);
		//getLastCudaError("LB_Kernel_Kum_New_Comp_SP_27 execution failed");
}

//////////////////////////////////////////////////////////////////////////
void CumulantOnePreconditionedErrorDiffusionChimCompSP27(unsigned int numberOfThreads,
																	real s9,
																	unsigned int* bcMatD,
																	unsigned int* neighborX,
																	unsigned int* neighborY,
																	unsigned int* neighborZ,
																	real* DD,
																	int size_Mat,
																	int size_Array,
																	int level,
																	real* forces,
																	bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	Cumulant_One_preconditioned_errorDiffusion_chim_Comp_SP_27 <<< grid.grid, grid.threads >>>(	s9,
																						bcMatD,
																						neighborX,
																						neighborY,
																						neighborZ,
																						DD,
																						size_Mat,
																						level,
																						forces,
																						EvenOrOdd);
		getLastCudaError("Cumulant_One_preconditioned_chim_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CumulantOnePreconditionedChimCompSP27(  unsigned int numberOfThreads,
														real s9,
														unsigned int* bcMatD,
														unsigned int* neighborX,
														unsigned int* neighborY,
														unsigned int* neighborZ,
														real* DD,
														int size_Mat,
														int size_Array,
														int level,
														real* forces,
														bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	Cumulant_One_preconditioned_chim_Comp_SP_27 <<< grid.grid, grid.threads >>>(	s9,
																		bcMatD,
																		neighborX,
																		neighborY,
																		neighborZ,
																		DD,
																		size_Mat,
																		level,
																		forces,
																		EvenOrOdd);
		getLastCudaError("Cumulant_One_preconditioned_chim_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CumulantOneChimCompSP27(unsigned int numberOfThreads,
										real s9,
										unsigned int* bcMatD,
										unsigned int* neighborX,
										unsigned int* neighborY,
										unsigned int* neighborZ,
										real* DD,
										int size_Mat,
										int size_Array,
										int level,
										real* forces,
										bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	Cumulant_One_chim_Comp_SP_27 <<< grid.grid, grid.threads >>>(	s9,
														bcMatD,
														neighborX,
														neighborY,
														neighborZ,
														DD,
														size_Mat,
														level,
														forces,
														EvenOrOdd);
		getLastCudaError("Cumulant_One_chim_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelKumIsoTestSP27(unsigned int numberOfThreads,
									 real s9,
									 unsigned int* bcMatD,
									 unsigned int* neighborX,
									 unsigned int* neighborY,
									 unsigned int* neighborZ,
									 real* DD,
									 real* dxxUx,
									 real* dyyUy,
									 real* dzzUz,
									 int size_Mat,
									 bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	LB_Kernel_Kum_IsoTest_SP_27<<< grid.grid, grid.threads >>>(s9,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													dxxUx,
													dyyUy,
													dzzUz,
													size_Mat,
													EvenOrOdd);
	getLastCudaError("LB_Kernel_Kum_IsoTest_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelKumCompSP27(  unsigned int numberOfThreads,
									real s9,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* DD,
									int size_Mat,
									bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


		LB_Kernel_Kum_Comp_SP_27<<< grid.grid, grid.threads >>>(s9,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													size_Mat,
													EvenOrOdd);
		getLastCudaError("LB_Kernel_Kum_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelPMCumOneCompSP27(unsigned int numberOfThreads,
									   real omega,
									   unsigned int* neighborX,
									   unsigned int* neighborY,
									   unsigned int* neighborZ,
									   real* DD,
									   int size_Mat,
									   int level,
									   real* forces,
									   real porosity,
									   real darcy,
									   real forchheimer,
									   unsigned int sizeOfPorousMedia,
									   unsigned int* nodeIdsPorousMedia,
									   bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	LB_Kernel_PM_Cum_One_Comp_SP_27 <<< grid.grid, grid.threads >>>(omega,
														  neighborX,
														  neighborY,
														  neighborZ,
														  DD,
														  size_Mat,
														  level,
														  forces,
														  porosity,
														  darcy,
														  forchheimer,
														  sizeOfPorousMedia,
														  nodeIdsPorousMedia,
														  EvenOrOdd);
	getLastCudaError("LB_Kernel_PM_Cum_One_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelWaleBySoniMalavCumAA2016CompSP27(
	unsigned int numberOfThreads,
	real s9,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int* neighborWSB,
	real* veloX,
	real* veloY,
	real* veloZ,
	real* DD,
	real* turbulentViscosity,
	int size_Mat,
	int size_Array,
	int level,
	real* forces,
	bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	LB_Kernel_WaleBySoniMalav_Cum_AA2016_Comp_SP_27 << < grid.grid, grid.threads >> >(
		s9,
		bcMatD,
		neighborX,
		neighborY,
		neighborZ,
		neighborWSB,
		veloX,
		veloY,
		veloZ,
		DD,
		turbulentViscosity,
		size_Mat,
		level,
		forces,
		EvenOrOdd);
	getLastCudaError("LB_Kernel_WaleBySoniMalav_Cum_AA2016_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelADincomp7(   unsigned int numberOfThreads,
								   real diffusivity,
								   unsigned int* bcMatD,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   real* DD,
								   real* DD7,
								   int size_Mat,
								   bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_AD_Incomp_7<<< grid.grid, grid.threads >>>( diffusivity,
												  bcMatD,
												  neighborX,
												  neighborY,
												  neighborZ,
												  DD,
												  DD7,
												  size_Mat,
												  EvenOrOdd);
      getLastCudaError("LB_Kernel_AD_Incomp_7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void KernelADincomp27( unsigned int numberOfThreads,
								  real diffusivity,
								  unsigned int* bcMatD,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  real* DD,
								  real* DD27,
								  int size_Mat,
								  bool EvenOrOdd)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LB_Kernel_AD_Incomp_27<<< grid.grid, grid.threads >>>( diffusivity,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													DD27,
													size_Mat,
													EvenOrOdd);
	getLastCudaError("LB_Kernel_AD_Incomp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void Init27( int myid,
                        int numprocs,
                        real u0,
                        unsigned int* geoD,
                        unsigned int* neighborX,
                        unsigned int* neighborY,
                        unsigned int* neighborZ,
                        real* vParab,
                        unsigned int size_Mat,
                        unsigned int grid_nx,
                        unsigned int grid_ny,
                        unsigned int grid_nz,
                        real* DD,
                        int level,
                        int maxlevel)
{
   dim3 threads       ( grid_nx, 1, 1 );
   dim3 grid          ( grid_ny, grid_nz );   // Gitter fuer Kollision und Propagation

	LBInit27<<< grid, threads >>> (  myid,
                                       numprocs,
                                       u0,
                                       geoD,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       vParab,
                                       size_Mat,
                                       grid_nx,
                                       grid_ny,
                                       grid_nz,
                                       DD,
                                       level,
                                       maxlevel);
	getLastCudaError("LBInit27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void InitNonEqPartSP27( unsigned int numberOfThreads,
                                   unsigned int* neighborX,
                                   unsigned int* neighborY,
                                   unsigned int* neighborZ,
                                   unsigned int* neighborWSB,
                                   unsigned int* geoD,
                                   real* rho,
                                   real* ux,
                                   real* uy,
                                   real* uz,
                                   unsigned int size_Mat,
                                   real* DD,
                                   real omega,
                                   bool EvenOrOdd)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBInitNonEqPartSP27<<< grid.grid, grid.threads >>>( neighborX,
                                                neighborY,
                                                neighborZ,
                                                neighborWSB,
                                                geoD,
                                                rho,
                                                ux,
                                                uy,
                                                uz,
                                                size_Mat,
                                                DD,
                                                omega,
                                                EvenOrOdd);
	getLastCudaError("LBInitNonEqPartSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void InitThS7(     unsigned int numberOfThreads,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int* geoD,
                              real* Conc,
                              real* ux,
                              real* uy,
                              real* uz,
                              unsigned int size_Mat,
                              real* DD7,
                              bool EvenOrOdd)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	InitAD7<<< grid.grid, grid.threads >>>( neighborX,
                                       neighborY,
                                       neighborZ,
                                       geoD,
                                       Conc,
                                       ux,
                                       uy,
                                       uz,
                                       size_Mat,
                                       DD7,
                                       EvenOrOdd);
	getLastCudaError("InitAD7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void InitADDev27( unsigned int numberOfThreads,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int* geoD,
                           real* Conc,
                           real* ux,
                           real* uy,
                           real* uz,
                           unsigned int size_Mat,
                           real* DD27,
                           bool EvenOrOdd)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	InitAD27<<< grid.grid, grid.threads >>>(neighborX,
                                       neighborY,
                                       neighborZ,
                                       geoD,
                                       Conc,
                                       ux,
                                       uy,
                                       uz,
                                       size_Mat,
                                       DD27,
                                       EvenOrOdd);
	getLastCudaError("InitAD27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void PostProcessorF3_2018Fehlberg(
	unsigned int numberOfThreads,
	real omega,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* rhoOut,
	real* vxOut,
	real* vyOut,
	real* vzOut,
	real* DDStart,
	real* G6,
	int size_Mat,
	int level,
	real* forces,
	bool EvenOrOdd)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LB_PostProcessor_F3_2018_Fehlberg <<< grid.grid, grid.threads >>> (   omega,
																  bcMatD,
																  neighborX,
																  neighborY,
																  neighborZ,
																  rhoOut,
																  vxOut,
																  vyOut,
																  vzOut,
																  DDStart,
																  G6,
																  size_Mat,
																  level,
																  forces,
																  EvenOrOdd);
	getLastCudaError("LB_PostProcessor_F3_2018_Fehlberg execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcMac27( real* vxD,
                           real* vyD,
                           real* vzD,
                           real* rhoD,
                           unsigned int* geoD,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int size_Mat,
                           unsigned int grid_nx,
                           unsigned int grid_ny,
                           unsigned int grid_nz,
                           real* DD,
                           bool isEvenTimestep)
{
   dim3 threads       ( grid_nx, 1, 1 );
   dim3 grid          ( grid_ny, grid_nz );

	LBCalcMac27<<< grid, threads >>> (  vxD,
                                          vyD,
                                          vzD,
                                          rhoD,
                                          geoD,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          DD,
                                          isEvenTimestep);
	getLastCudaError("LBCalcMac27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcMacSP27( real* vxD,
                             real* vyD,
                             real* vzD,
                             real* rhoD,
                             real* pressD,
                             unsigned int* geoD,
                             unsigned int* neighborX,
                             unsigned int* neighborY,
                             unsigned int* neighborZ,
                             unsigned int size_Mat,
                             unsigned int numberOfThreads,
                             real* DD,
                             bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcMacSP27<<< grid.grid, grid.threads >>> (   vxD,
                                             vyD,
                                             vzD,
                                             rhoD,
                                             pressD,
                                             geoD,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             DD,
                                             isEvenTimestep);
	getLastCudaError("LBCalcMacSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcMacCompSP27( real* vxD,
								 real* vyD,
								 real* vzD,
								 real* rhoD,
								 real* pressD,
								 unsigned int* geoD,
								 unsigned int* neighborX,
								 unsigned int* neighborY,
								 unsigned int* neighborZ,
								 unsigned int size_Mat,
								 unsigned int numberOfThreads,
								 real* DD,
								 bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcMacCompSP27<<< grid.grid, grid.threads >>> (   vxD,
												 vyD,
												 vzD,
												 rhoD,
												 pressD,
												 geoD,
												 neighborX,
												 neighborY,
												 neighborZ,
												 size_Mat,
												 DD,
												 isEvenTimestep);
	getLastCudaError("LBCalcMacSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcMacThS7(  real* Conc,
                              unsigned int* geoD,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              unsigned int numberOfThreads,
                              real* DD7,
                              bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	CalcConc7<<< grid.grid, grid.threads >>> (Conc,
                                          geoD,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          DD7,
                                          isEvenTimestep);
	getLastCudaError("CalcConc7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void PlaneConcThS7(real* Conc,
							  int* kPC,
							  unsigned int numberOfPointskPC,
							  unsigned int* geoD,
							  unsigned int* neighborX,
							  unsigned int* neighborY,
							  unsigned int* neighborZ,
							  unsigned int size_Mat,
                              unsigned int numberOfThreads,
							  real* DD7,
							  bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfPointskPC);

	GetPlaneConc7<<< grid.grid, grid.threads >>> (	Conc,
												kPC,
												numberOfPointskPC,
												geoD,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												DD7,
												isEvenTimestep);
	getLastCudaError("GetPlaneConc7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void PlaneConcThS27(real* Conc,
							   int* kPC,
							   unsigned int numberOfPointskPC,
							   unsigned int* geoD,
							   unsigned int* neighborX,
							   unsigned int* neighborY,
							   unsigned int* neighborZ,
							   unsigned int size_Mat,
                               unsigned int numberOfThreads,
							   real* DD27,
							   bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfPointskPC);

	GetPlaneConc27<<< grid.grid, grid.threads >>> (	Conc,
												kPC,
												numberOfPointskPC,
												geoD,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												DD27,
												isEvenTimestep);
	getLastCudaError("GetPlaneConc27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcConcentration27( unsigned int numberOfThreads,
                                     real* Conc,
                                     unsigned int* geoD,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     real* DD27,
                                     bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	CalcConc27<<< grid.grid, grid.threads >>> (  Conc,
                                             geoD,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             DD27,
                                             isEvenTimestep);
	getLastCudaError("CalcConc27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcMedSP27(  real* vxD,
                              real* vyD,
                              real* vzD,
                              real* rhoD,
                              real* pressD,
                              unsigned int* geoD,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              unsigned int numberOfThreads,
                              real* DD,
                              bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcMedSP27<<< grid.grid, grid.threads >>> (   vxD,
                                             vyD,
                                             vzD,
                                             rhoD,
                                             pressD,
                                             geoD,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             DD,
                                             isEvenTimestep);
	getLastCudaError("LBCalcMedSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcMedCompSP27(  real* vxD,
								  real* vyD,
								  real* vzD,
								  real* rhoD,
								  real* pressD,
								  unsigned int* geoD,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  unsigned int size_Mat,
								  unsigned int numberOfThreads,
								  real* DD,
								  bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcMedCompSP27<<< grid.grid, grid.threads >>> (   vxD,
												 vyD,
												 vzD,
												 rhoD,
												 pressD,
												 geoD,
												 neighborX,
												 neighborY,
												 neighborZ,
												 size_Mat,
												 DD,
												 isEvenTimestep);
	getLastCudaError("LBCalcMedSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcMedCompAD27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	real* concD,
	unsigned int* geoD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	unsigned int numberOfThreads,
	real* DD,
	real* DD_AD,
	bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcMedCompAD27 <<< grid.grid, grid.threads >>> (
		vxD,
		vyD,
		vzD,
		rhoD,
		pressD,
		concD,
		geoD,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		DD,
		DD_AD,
		isEvenTimestep);
	getLastCudaError("LBCalcMedAD27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcMacMedSP27(  real* vxD,
                                 real* vyD,
                                 real* vzD,
                                 real* rhoD,
                                 real* pressD,
                                 unsigned int* geoD,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 unsigned int tdiff,
                                 unsigned int size_Mat,
                                 unsigned int numberOfThreads,
                                 bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcMacMedSP27<<< grid.grid, grid.threads >>> (   vxD,
                                                vyD,
                                                vzD,
                                                rhoD,
                                                pressD,
                                                geoD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                tdiff,
                                                size_Mat,
                                                isEvenTimestep);
	getLastCudaError("LBCalcMacMedSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ResetMedianValuesSP27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	unsigned int size_Mat,
	unsigned int numberOfThreads,
	bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	LBResetMedianValuesSP27 << < grid.grid, grid.threads >> > (
		vxD,
		vyD,
		vzD,
		rhoD,
		pressD,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("LBResetMedianValuesSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ResetMedianValuesAD27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	real* concD,
	unsigned int size_Mat,
	unsigned int numberOfThreads,
	bool isEvenTimestep)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBResetMedianValuesAD27 << < grid.grid, grid.threads >> > (
		vxD,
		vyD,
		vzD,
		rhoD,
		pressD,
		concD,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("LBResetMedianValuesAD27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void Calc2ndMomentsIncompSP27(real* kxyFromfcNEQ,
										 real* kyzFromfcNEQ,
										 real* kxzFromfcNEQ,
										 real* kxxMyyFromfcNEQ,
										 real* kxxMzzFromfcNEQ,
										 unsigned int* geoD,
										 unsigned int* neighborX,
										 unsigned int* neighborY,
										 unsigned int* neighborZ,
										 unsigned int size_Mat,
										 unsigned int numberOfThreads,
										 real* DD,
										 bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalc2ndMomentsIncompSP27<<< grid.grid, grid.threads >>> (  kxyFromfcNEQ,
														 kyzFromfcNEQ,
														 kxzFromfcNEQ,
														 kxxMyyFromfcNEQ,
														 kxxMzzFromfcNEQ,
														 geoD,
														 neighborX,
														 neighborY,
														 neighborZ,
														 size_Mat,
														 DD,
														 isEvenTimestep);
	getLastCudaError("LBCalc2ndMomentsIncompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void Calc2ndMomentsCompSP27( real* kxyFromfcNEQ,
										real* kyzFromfcNEQ,
										real* kxzFromfcNEQ,
										real* kxxMyyFromfcNEQ,
										real* kxxMzzFromfcNEQ,
										unsigned int* geoD,
										unsigned int* neighborX,
										unsigned int* neighborY,
										unsigned int* neighborZ,
										unsigned int size_Mat,
										unsigned int numberOfThreads,
										real* DD,
										bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalc2ndMomentsCompSP27<<< grid.grid, grid.threads >>> (kxyFromfcNEQ,
													 kyzFromfcNEQ,
													 kxzFromfcNEQ,
													 kxxMyyFromfcNEQ,
													 kxxMzzFromfcNEQ,
													 geoD,
													 neighborX,
													 neighborY,
													 neighborZ,
													 size_Mat,
													 DD,
													 isEvenTimestep);
	getLastCudaError("LBCalc2ndMomentsCompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void Calc3rdMomentsIncompSP27(real* CUMbbb,
										 real* CUMabc,
										 real* CUMbac,
										 real* CUMbca,
										 real* CUMcba,
										 real* CUMacb,
										 real* CUMcab,
										 unsigned int* geoD,
										 unsigned int* neighborX,
										 unsigned int* neighborY,
										 unsigned int* neighborZ,
										 unsigned int size_Mat,
										 unsigned int numberOfThreads,
										 real* DD,
										 bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalc3rdMomentsIncompSP27<<< grid.grid, grid.threads >>> (  CUMbbb,
														 CUMabc,
														 CUMbac,
														 CUMbca,
														 CUMcba,
														 CUMacb,
														 CUMcab,
														 geoD,
														 neighborX,
														 neighborY,
														 neighborZ,
														 DD,
														 size_Mat,
														 isEvenTimestep);
	getLastCudaError("LBCalc3rdMomentsIncompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void Calc3rdMomentsCompSP27( real* CUMbbb,
										real* CUMabc,
										real* CUMbac,
										real* CUMbca,
										real* CUMcba,
										real* CUMacb,
										real* CUMcab,
										unsigned int* geoD,
										unsigned int* neighborX,
										unsigned int* neighborY,
										unsigned int* neighborZ,
										unsigned int size_Mat,
										unsigned int numberOfThreads,
										real* DD,
										bool isEvenTimestep)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalc3rdMomentsCompSP27<<< grid.grid, grid.threads >>> (CUMbbb,
													 CUMabc,
													 CUMbac,
													 CUMbca,
													 CUMcba,
													 CUMacb,
													 CUMcab,
													 geoD,
													 neighborX,
													 neighborY,
													 neighborZ,
													 DD,
													 size_Mat,
													 isEvenTimestep);
	getLastCudaError("LBCalc3rdMomentsCompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcHigherMomentsIncompSP27(real* CUMcbb,
											real* CUMbcb,
											real* CUMbbc,
											real* CUMcca,
											real* CUMcac,
											real* CUMacc,
											real* CUMbcc,
											real* CUMcbc,
											real* CUMccb,
											real* CUMccc,
											unsigned int* geoD,
											unsigned int* neighborX,
											unsigned int* neighborY,
											unsigned int* neighborZ,
											unsigned int size_Mat,
											unsigned int numberOfThreads,
											real* DD,
											bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcHigherMomentsIncompSP27<<< grid.grid, grid.threads >>> (CUMcbb,
														  CUMbcb,
														  CUMbbc,
														  CUMcca,
														  CUMcac,
														  CUMacc,
														  CUMbcc,
														  CUMcbc,
														  CUMccb,
														  CUMccc,
														  geoD,
														  neighborX,
														  neighborY,
														  neighborZ,
														  DD,
														  size_Mat,
														  isEvenTimestep);
	getLastCudaError("LBCalcHigherMomentsIncompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcHigherMomentsCompSP27(  real* CUMcbb,
											real* CUMbcb,
											real* CUMbbc,
											real* CUMcca,
											real* CUMcac,
											real* CUMacc,
											real* CUMbcc,
											real* CUMcbc,
											real* CUMccb,
											real* CUMccc,
											unsigned int* geoD,
											unsigned int* neighborX,
											unsigned int* neighborY,
											unsigned int* neighborZ,
											unsigned int size_Mat,
											unsigned int numberOfThreads,
											real* DD,
											bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcHigherMomentsCompSP27<<< grid.grid, grid.threads >>> (  CUMcbb,
														  CUMbcb,
														  CUMbbc,
														  CUMcca,
														  CUMcac,
														  CUMacc,
														  CUMbcc,
														  CUMcbc,
														  CUMccb,
														  CUMccc,
														  geoD,
														  neighborX,
														  neighborY,
														  neighborZ,
														  DD,
														  size_Mat,
														  isEvenTimestep);
	getLastCudaError("LBCalcHigherMomentsCompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void LBCalcMeasurePoints27(real* vxMP,
                                      real* vyMP,
                                      real* vzMP,
                                      real* rhoMP,
                                      unsigned int* kMP,
                                      unsigned int numberOfPointskMP,
                                      unsigned int MPClockCycle,
                                      unsigned int t,
                                      unsigned int* geoD,
                                      unsigned int* neighborX,
                                      unsigned int* neighborY,
                                      unsigned int* neighborZ,
                                      unsigned int size_Mat,
                                      real* DD,
                                      unsigned int numberOfThreads,
                                      bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfPointskMP);

	LBCalcMeasurePoints<<< grid.grid, grid.threads >>> (vxMP,
                                                vyMP,
                                                vzMP,
                                                rhoMP,
                                                kMP,
                                                numberOfPointskMP,
                                                MPClockCycle,
                                                t,
                                                geoD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                DD,
                                                isEvenTimestep);
	getLastCudaError("LBCalcMeasurePoints execution failed");
}
//////////////////////////////////////////////////////////////////////////
void BcPress27( int nx,
                           int ny,
                           int tz,
                           unsigned int grid_nx,
                           unsigned int grid_ny,
                           unsigned int* bcMatD,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           real* DD,
                           unsigned int size_Mat,
                           bool isEvenTimestep)
{
	dim3 threads       ( grid_nx, 1, 1 );
	dim3 grid          ( grid_ny, 1 );

	LB_BC_Press_East27<<< grid, threads >>> ( nx,
                                                ny,
                                                tz,
                                                bcMatD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                DD,
                                                size_Mat,
                                                isEvenTimestep);
	getLastCudaError("LB_BC_Press_East27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void BcVel27(int nx,
                        int ny,
                        int nz,
                        int itz,
                        unsigned int grid_nx,
                        unsigned int grid_ny,
                        unsigned int* bcMatD,
                        unsigned int* neighborX,
                        unsigned int* neighborY,
                        unsigned int* neighborZ,
                        real* DD,
                        unsigned int size_Mat,
                        bool isEvenTimestep,
                        real u0x,
                        real om)
{
	dim3 threads       ( grid_nx, 1, 1 );
	dim3 grid          ( grid_ny, 1 );

	LB_BC_Vel_West_27<<< grid, threads >>> (  nx,
                                                ny,
                                                nz,
                                                itz,
                                                bcMatD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                DD,
                                                size_Mat,
                                                isEvenTimestep,
                                                u0x,
                                                grid_nx,
                                                grid_ny,
                                                om);
	getLastCudaError("LB_BC_Vel_West_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADPressDev7( unsigned int numberOfThreads,
                              real* DD,
                              real* DD7,
                              real* temp,
                              real* velo,
                              real diffusivity,
                              int* k_Q,
                              real* QQ,
                              unsigned int numberOfBCnodes,
                              real om1,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADPress7<<< grid.grid, grid.threads >>>( DD,
                                       DD7,
                                       temp,
                                       velo,
                                       diffusivity,
                                       k_Q,
                                       QQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
	getLastCudaError("QADPress7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADPressDev27(unsigned int numberOfThreads,
                              real* DD,
                              real* DD27,
                              real* temp,
                              real* velo,
                              real diffusivity,
                              int* k_Q,
                              real* QQ,
                              unsigned int numberOfBCnodes,
                              real om1,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADPress27<<< grid.grid, grid.threads >>>(   DD,
                                          DD27,
                                          temp,
                                          velo,
                                          diffusivity,
                                          k_Q,
                                          QQ,
                                          numberOfBCnodes,
                                          om1,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          isEvenTimestep);
	getLastCudaError("QADPress27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADPressNEQNeighborDev27(
											unsigned int numberOfThreads,
											real* DD,
											real* DD27,
											int* k_Q,
											int* k_N,
											int numberOfBCnodes,
											unsigned int* neighborX,
											unsigned int* neighborY,
											unsigned int* neighborZ,
											unsigned int size_Mat,
											bool isEvenTimestep
										)
{

	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

	QADPressNEQNeighbor27<<< grid.grid, grid.threads >>>(
												DD,
												DD27,
												k_Q,
												k_N,
												numberOfBCnodes,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep
											  );
   	getLastCudaError("QADPressNEQNeighbor27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADVelDev7(unsigned int numberOfThreads,
                           real* DD,
                           real* DD7,
                           real* temp,
                           real* velo,
                           real diffusivity,
                           int* k_Q,
                           real* QQ,
                           unsigned int numberOfBCnodes,
                           real om1,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int size_Mat,
                           bool isEvenTimestep)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADVel7<<< grid.grid, grid.threads >>> (  
                                       DD,
                                       DD7,
                                       temp,
                                       velo,
                                       diffusivity,
                                       k_Q,
                                       QQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
	getLastCudaError("QADVel7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADVelDev27(  unsigned int numberOfThreads,
                              real* DD,
                              real* DD27,
                              real* temp,
                              real* velo,
                              real diffusivity,
                              int* k_Q,
                              real* QQ,
                              unsigned int numberOfBCnodes,
                              real om1,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADVel27<<< grid.grid, grid.threads >>> ( DD,
                                      DD27,
                                      temp,
                                      velo,
                                      diffusivity,
                                      k_Q,
                                      QQ,
                                      numberOfBCnodes,
                                      om1,
                                      neighborX,
                                      neighborY,
                                      neighborZ,
                                      size_Mat,
                                      isEvenTimestep);
      getLastCudaError("QADVel27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADDev7(unsigned int numberOfThreads,
                        real* DD,
                        real* DD7,
                        real* temp,
                        real diffusivity,
                        int* k_Q,
                        real* QQ,
                        unsigned int numberOfBCnodes,
                        real om1,
                        unsigned int* neighborX,
                        unsigned int* neighborY,
                        unsigned int* neighborZ,
                        unsigned int size_Mat,
                        bool isEvenTimestep)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QAD7<<< grid.grid, grid.threads >>> (     DD,
                                       DD7,
                                       temp,
                                       diffusivity,
                                       k_Q,
                                       QQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
      getLastCudaError("QAD7 execution failed");
}


//////////////////////////////////////////////////////////////////////////
// Other advection diffusion kernels are in kernel factory :(
void FactorizedCentralMomentsAdvectionDiffusionDeviceKernel(
   uint numberOfThreads,
   real omegaDiffusivity,
   uint* typeOfGridNode,
   uint* neighborX,
   uint* neighborY,
   uint* neighborZ,
   real* distributions,
   real* distributionsAD,
   int size_Mat,
   real* forces,
   bool isEvenTimestep)
{
   int Grid = (size_Mat / numberOfThreads) + 1;
   dim3 grid(Grid, 1, 1);
   dim3 threads(numberOfThreads, 1, 1);

   Factorized_Central_Moments_Advection_Diffusion_Device_Kernel <<< grid, threads >>> (
      omegaDiffusivity,
      typeOfGridNode,
      neighborX,
      neighborY,
      neighborZ,
      distributions,
      distributionsAD,
      size_Mat,
      forces,
      isEvenTimestep);
   getLastCudaError("Factorized_Central_Moments_Advection_Diffusion_Device_Kernel execution failed");
}

//////////////////////////////////////////////////////////////////////////
void ADSlipVelDevComp(
	uint numberOfThreads,
	real * normalX,
	real * normalY,
	real * normalZ,
	real * distributions,
	real * distributionsAD,
	int* QindexArray,
	real * Qarrays,
	uint numberOfBCnodes,
	real omegaDiffusivity,
	uint * neighborX,
	uint * neighborY,
	uint * neighborZ,
	uint size_Mat,
	bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

	AD_SlipVelDeviceComp << < grid.grid, grid.threads >> > (
		normalX,
		normalY,
		normalZ,
		distributions,
		distributionsAD,
		QindexArray,
		Qarrays,
		numberOfBCnodes,
		omegaDiffusivity,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("AD_SlipVelDeviceComp execution failed");
}
//////////////////////////////////////////////////////////////////////////

void QADDirichletDev27( unsigned int numberOfThreads,
								   real* DD,
								   real* DD27,
								   real* temp,
								   real diffusivity,
								   int* k_Q,
								   real* QQ,
								   unsigned int numberOfBCnodes,
								   real om1,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep)
{
   	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADDirichlet27<<< grid.grid, grid.threads >>> (
											   DD,
											   DD27,
											   temp,
											   diffusivity,
											   k_Q,
											   QQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QADDirichletDev27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADBBDev27(unsigned int numberOfThreads,
                           real* DD,
                           real* DD27,
                           real* temp,
                           real diffusivity,
                           int* k_Q,
                           real* QQ,
                           unsigned int numberOfBCnodes,
                           real om1,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int size_Mat,
                           bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADBB27<<< grid.grid, grid.threads >>> (  DD,
                                       DD27,
                                       temp,
                                       diffusivity,
                                       k_Q,
                                       QQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
      getLastCudaError("QADBB27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QNoSlipADincompDev7(unsigned int numberOfThreads,
									real* DD,
									real* DD7,
									real* temp,
									real diffusivity,
									int* k_Q,
									real* QQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QNoSlipADincomp7<<< grid.grid, grid.threads >>> (
											   DD,
											   DD7,
											   temp,
											   diffusivity,
											   k_Q,
											   QQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QNoSlipADincomp7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QNoSlipADincompDev27(  unsigned int numberOfThreads,
									   real* DD,
									   real* DD27,
									   real* temp,
									   real diffusivity,
									   int* k_Q,
									   real* QQ,
									   unsigned int numberOfBCnodes,
									   real om1,
									   unsigned int* neighborX,
									   unsigned int* neighborY,
									   unsigned int* neighborZ,
									   unsigned int size_Mat,
									   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QNoSlipADincomp27<<< grid.grid, grid.threads >>> (
											   DD,
											   DD27,
											   temp,
											   diffusivity,
											   k_Q,
											   QQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QNoSlipADincomp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADVeloIncompDev7( unsigned int numberOfThreads,
								   real* DD,
								   real* DD7,
								   real* temp,
								   real* velo,
								   real diffusivity,
								   int* k_Q,
								   real* QQ,
								   unsigned int numberOfBCnodes,
								   real om1,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADVeloIncomp7<<< grid.grid, grid.threads >>> ( DD,
	  										   DD7,
											   temp,
											   velo,
											   diffusivity,
											   k_Q,
											   QQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QADVeloIncomp7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADVeloIncompDev27(   unsigned int numberOfThreads,
									  real* DD,
									  real* DD27,
									  real* temp,
									  real* velo,
									  real diffusivity,
									  int* k_Q,
									  real* QQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADVeloIncomp27<<< grid.grid, grid.threads >>> (
											  DD,
											  DD27,
											  temp,
											  velo,
											  diffusivity,
											  k_Q,
											  QQ,
											  numberOfBCnodes,
											  om1,
											  neighborX,
											  neighborY,
											  neighborZ,
											  size_Mat,
											  isEvenTimestep);
      getLastCudaError("QADVeloIncomp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADPressIncompDev7( unsigned int numberOfThreads,
									  real* DD,
									  real* DD7,
									  real* temp,
									  real* velo,
									  real diffusivity,
									  int* k_Q,
									  real* QQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADPressIncomp7<<< grid.grid, grid.threads >>>(
											   DD,
											   DD7,
											   temp,
											   velo,
											   diffusivity,
											   k_Q,
											   QQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QADPressIncomp7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QADPressIncompDev27(  unsigned int numberOfThreads,
									  real* DD,
									  real* DD27,
									  real* temp,
									  real* velo,
									  real diffusivity,
									  int* k_Q,
									  real* QQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QADPressIncomp27<<< grid.grid, grid.threads >>>(DD, 
	  										  DD27, 
											  temp,
											  velo,
											  diffusivity,
											  k_Q,
											  QQ,
											  numberOfBCnodes,
											  om1,
											  neighborX,
											  neighborY,
											  neighborZ,
											  size_Mat,
											  isEvenTimestep);
      getLastCudaError("QADPressIncomp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QDev27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

      QDevice27<<< grid, threads >>> (
            parameterDevice->distributions.f[0],
            boundaryCondition->k,
            boundaryCondition->q27[0],
            boundaryCondition->numberOfBCnodes,
            parameterDevice->omega,
            parameterDevice->neighborX,
            parameterDevice->neighborY,
            parameterDevice->neighborZ,
            parameterDevice->numberOfNodes,
            parameterDevice->isEvenTimestep);

      getLastCudaError("QDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QDevComp27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

      QDeviceComp27<<< grid, threads >>> (
           parameterDevice->distributions.f[0],
           boundaryCondition->k,
           boundaryCondition->q27[0],
           boundaryCondition->numberOfBCnodes,
           parameterDevice->omega,
           parameterDevice->neighborX,
           parameterDevice->neighborY,
           parameterDevice->neighborZ,
           parameterDevice->numberOfNodes,
           parameterDevice->isEvenTimestep);
      getLastCudaError("QDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QDevCompThinWalls27(unsigned int numberOfThreads,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* geom,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int* neighborWSB,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QDeviceCompThinWallsPartOne27 <<< grid.grid, grid.threads >>> (DD,
														 k_Q,
														 QQ,
														 numberOfBCnodes,
														 om1,
														 neighborX,
														 neighborY,
														 neighborZ,
														 size_Mat,
														 isEvenTimestep);
   getLastCudaError("QDeviceCompThinWallsPartOne27 execution failed");

   QThinWallsPartTwo27 <<< grid.grid, grid.threads >>> ( DD,
												k_Q,
												QQ,
												numberOfBCnodes,
												geom,
												neighborX,
												neighborY,
												neighborZ,
												neighborWSB,
												size_Mat,
												isEvenTimestep);
   getLastCudaError("QThinWallsPartTwo27 execution failed");

}
//////////////////////////////////////////////////////////////////////////
void QDev3rdMomentsComp27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1);

   QDevice3rdMomentsComp27<<< grid, threads >>> (
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("QDevice3rdMomentsComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QDevIncompHighNu27( unsigned int numberOfThreads,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QDeviceIncompHighNu27<<< grid.grid, grid.threads >>> (
												   DD,
												   k_Q,
												   QQ,
												   numberOfBCnodes,
												   om1,
												   neighborX,
												   neighborY,
												   neighborZ,
												   size_Mat,
												   isEvenTimestep);
      getLastCudaError("QDeviceIncompHighNu27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QDevCompHighNu27(   unsigned int numberOfThreads,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QDeviceCompHighNu27<<< grid.grid, grid.threads >>> (
												   DD,
												   k_Q,
												   QQ,
												   numberOfBCnodes,
												   om1,
												   neighborX,
												   neighborY,
												   neighborZ,
												   size_Mat,
												   isEvenTimestep);
      getLastCudaError("QDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDevicePlainBB27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QVelDevPlainBB27<<< grid, threads >>> (
         boundaryCondition->Vx,
         boundaryCondition->Vy,
         boundaryCondition->Vz,
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("QVelDevicePlainBB27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDeviceCouette27(unsigned int numberOfThreads,
									real* vx,
									real* vy,
									real* vz,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QVelDevCouette27<<< grid.grid, grid.threads >>> ( vx,
												vy,
												vz,
												DD,
												k_Q,
												QQ,
												numberOfBCnodes,
												om1,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep);
      getLastCudaError("QVelDevicePlainBB27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDevice1h27(   unsigned int numberOfThreads,
								  int nx,
								  int ny,
								  real* vx,
								  real* vy,
								  real* vz,
								  real* DD,
								  int* k_Q,
								  real* QQ,
								  unsigned int numberOfBCnodes,
								  real om1,
								  real Phi,
								  real angularVelocity,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  real* coordX,
								  real* coordY,
								  real* coordZ,
								  unsigned int size_Mat,
								  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDev1h27<<< grid.grid, grid.threads >>> (nx,
                                          ny,
                                          vx,
                                          vy,
                                          vz,
                                          DD,
                                          k_Q,
                                          QQ,
                                          numberOfBCnodes,
                                          om1,
										  Phi,
										  angularVelocity,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
										  coordX,
										  coordY,
										  coordZ,
                                          size_Mat,
                                          isEvenTimestep);
      getLastCudaError("QVelDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDev27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

      QVelDevice27<<< grid, threads >>> (
            parameterDevice->nx,
            parameterDevice->ny,
            boundaryCondition->Vx,
            boundaryCondition->Vy,
            boundaryCondition->Vz,
            parameterDevice->distributions.f[0],
            boundaryCondition->k,
            boundaryCondition->q27[0],
            boundaryCondition->numberOfBCnodes,
            parameterDevice->omega,
            parameterDevice->neighborX,
            parameterDevice->neighborY,
            parameterDevice->neighborZ,
            parameterDevice->numberOfNodes,
            parameterDevice->isEvenTimestep);
      getLastCudaError("QVelDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDevCompPlusSlip27(unsigned int numberOfThreads,
									  real* vx,
									  real* vy,
									  real* vz,
									  real* DD,
									  int* k_Q,
									  real* QQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QVelDeviceCompPlusSlip27<<< grid.grid, grid.threads >>> (
													  vx,
													  vy,
													  vz,
													  DD,
													  k_Q,
													  QQ,
													  numberOfBCnodes,
													  om1,
													  neighborX,
													  neighborY,
													  neighborZ,
													  size_Mat,
													  isEvenTimestep);
      getLastCudaError("QVelDeviceCompPlusSlip27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDevComp27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid(parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QVelDeviceComp27<<< grid, threads >>> (
            boundaryCondition->Vx,
            boundaryCondition->Vy,
            boundaryCondition->Vz,
            parameterDevice->distributions.f[0],
            boundaryCondition->k,        
            boundaryCondition->q27[0],
            boundaryCondition->numberOfBCnodes,
            parameterDevice->omega,
            parameterDevice->neighborX,
            parameterDevice->neighborY,
            parameterDevice->neighborZ,
            parameterDevice->numberOfNodes,
            parameterDevice->isEvenTimestep);
   getLastCudaError("QVelDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDevCompThinWalls27(unsigned int numberOfThreads,
							           real* vx,
							           real* vy,
							           real* vz,
							           real* DD,
							           int* k_Q,
							           real* QQ,
							           unsigned int numberOfBCnodes,
							           real om1,
									     unsigned int* geom,
							           unsigned int* neighborX,
							           unsigned int* neighborY,
							           unsigned int* neighborZ,
									     unsigned int* neighborWSB,
							           unsigned int size_Mat,
							           bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDeviceCompThinWallsPartOne27<<< grid.grid, grid.threads >>> (vx,
											                  vy,
											                  vz,
											                  DD,
											                  k_Q,
											                  QQ,
											                  numberOfBCnodes,
											                  om1,
											                  neighborX,
											                  neighborY,
											                  neighborZ,
											                  size_Mat,
											                  isEvenTimestep);
   getLastCudaError("QVelDeviceCompThinWallsPartOne27 execution failed");

	QThinWallsPartTwo27 <<< grid.grid, grid.threads >>> (
       DD,
       k_Q,
       QQ,
       numberOfBCnodes,
       geom,
       neighborX,
       neighborY,
       neighborZ,
       neighborWSB,
       size_Mat,
       isEvenTimestep);
   getLastCudaError("QThinWallsPartTwo27 execution failed");
}

void QVelDevCompZeroPress27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QVelDeviceCompZeroPress27<<< grid, threads >>> (
            boundaryCondition->Vx,
            boundaryCondition->Vy,
            boundaryCondition->Vz,
            parameterDevice->distributions.f[0],
            boundaryCondition->k,
            boundaryCondition->q27[0],
            boundaryCondition->numberOfBCnodes,
            parameterDevice->omega,
            parameterDevice->neighborX,
            parameterDevice->neighborY,
            parameterDevice->neighborZ,
            parameterDevice->numberOfNodes,
            parameterDevice->isEvenTimestep);
   getLastCudaError("QVelDeviceCompZeroPress27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDevIncompHighNu27(unsigned int numberOfThreads,
									  real* vx,
									  real* vy,
									  real* vz,
									  real* DD,
									  int* k_Q,
									  real* QQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QVelDeviceIncompHighNu27<<< grid.grid, grid.threads >>> (
													  vx,
													  vy,
													  vz,
													  DD,
													  k_Q,
													  QQ,
													  numberOfBCnodes,
													  om1,
													  neighborX,
													  neighborY,
													  neighborZ,
													  size_Mat,
													  isEvenTimestep);
      getLastCudaError("QVelDeviceIncompHighNu27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVelDevCompHighNu27(  unsigned int numberOfThreads,
									  real* vx,
									  real* vy,
									  real* vz,
									  real* DD,
									  int* k_Q,
									  real* QQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      QVelDeviceCompHighNu27<<< grid.grid, grid.threads >>> (
													  vx,
													  vy,
													  vz,
													  DD,
													  k_Q,
													  QQ,
													  numberOfBCnodes,
													  om1,
													  neighborX,
													  neighborY,
													  neighborZ,
													  size_Mat,
													  isEvenTimestep);
      getLastCudaError("QVelDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVeloDevEQ27(unsigned int numberOfThreads,
							 real* VeloX,
							 real* VeloY,
							 real* VeloZ,
							 real* DD,
							 int* k_Q,
							 int numberOfBCnodes,
							 real om1,
							 unsigned int* neighborX,
							 unsigned int* neighborY,
							 unsigned int* neighborZ,
							 unsigned int size_Mat,
							 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVeloDeviceEQ27<<< grid.grid, grid.threads >>> (VeloX,
											 VeloY,
											 VeloZ,
											 DD,
											 k_Q,
											 numberOfBCnodes,
											 om1,
											 neighborX,
											 neighborY,
											 neighborZ,
											 size_Mat,
											 isEvenTimestep);
      getLastCudaError("QVeloDeviceEQ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QVeloStreetDevEQ27(
	uint  numberOfThreads,
	real* veloXfraction,
	real* veloYfraction,
	int*  naschVelo,
	real* DD,
	int*  naschIndex,
	int   numberOfStreetNodes,
	real  velocityRatio,
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	uint  size_Mat,
	bool  isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfStreetNodes);

	QVeloStreetDeviceEQ27 << < grid.grid, grid.threads >> > (
		veloXfraction,
		veloYfraction,
		naschVelo,
		DD,
		naschIndex,
		numberOfStreetNodes,
		velocityRatio,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("QVeloStreetDeviceEQ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QSlipDev27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads, boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QSlipDevice27<<< grid, threads >>> (
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("QSlipDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QSlipDevCompTurbulentViscosity27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads, boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );
   
   QSlipDeviceComp27TurbViscosity<<< grid, threads >>> (
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->turbViscosity,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("QSlipDeviceComp27TurbViscosity execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QSlipPressureDevCompTurbulentViscosity27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads, boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QSlipPressureDeviceComp27TurbViscosity<<< grid, threads >>> (
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->turbViscosity,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("QSlipDeviceComp27TurbViscosity execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QSlipDevComp27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads, boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );
   
   QSlipDeviceComp27<<< grid, threads >>> (
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("QSlipDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void BBSlipDevComp27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads, boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QSlipDeviceComp27<<< grid, threads >>> (
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("BBSlipDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QSlipGeomDevComp27(unsigned int numberOfThreads,
								   real* DD,
								   int* k_Q,
								   real* QQ,
								   unsigned int numberOfBCnodes,
								   real om1,
								   real* NormalX,
								   real* NormalY,
								   real* NormalZ,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep)
{
	vf::cuda::CudaGrid grid(numberOfThreads, numberOfBCnodes);

   QSlipGeomDeviceComp27<<< grid.grid, grid.threads >>> (DD,
												   k_Q,
												   QQ,
												   numberOfBCnodes,
												   om1,
												   NormalX,
												   NormalY,
												   NormalZ,
												   neighborX,
												   neighborY,
												   neighborZ,
												   size_Mat,
												   isEvenTimestep);
   getLastCudaError("QSlipGeomDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QSlipNormDevComp27(unsigned int numberOfThreads,
								   real* DD,
								   int* k_Q,
								   real* QQ,
								   unsigned int numberOfBCnodes,
								   real om1,
								   real* NormalX,
								   real* NormalY,
								   real* NormalZ,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QSlipNormDeviceComp27<<< grid.grid, grid.threads >>> (DD,
												   k_Q,
												   QQ,
												   numberOfBCnodes,
												   om1,
												   NormalX,
												   NormalY,
												   NormalZ,
												   neighborX,
												   neighborY,
												   neighborZ,
												   size_Mat,
												   isEvenTimestep);
      getLastCudaError("QSlipGeomDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QStressDevComp27(Parameter *para,  QforBoundaryConditions* boundaryCondition, const int level)
{
   dim3 grid = vf::cuda::getCudaGrid(  para->getParD(level)->numberofthreads, boundaryCondition->numberOfBCnodes);
   dim3 threads(para->getParD(level)->numberofthreads, 1, 1 );

      QStressDeviceComp27<<< grid, threads >>> (
         para->getParD(level)->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->kN,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         para->getParD(level)->omega,
         para->getParD(level)->turbViscosity,
         para->getParD(level)->velocityX,
         para->getParD(level)->velocityY,
         para->getParD(level)->velocityY,
         boundaryCondition->normalX,
         boundaryCondition->normalY,
         boundaryCondition->normalZ,
         boundaryCondition->Vx,
         boundaryCondition->Vy,
         boundaryCondition->Vz,
         boundaryCondition->Vx1,
         boundaryCondition->Vy1,
         boundaryCondition->Vz1,
         para->getParD(level)->wallModel.samplingOffset,
         para->getParD(level)->wallModel.z0,
         para->getHasWallModelMonitor(),
         para->getParD(level)->wallModel.u_star,
         para->getParD(level)->wallModel.Fx,
         para->getParD(level)->wallModel.Fy,
         para->getParD(level)->wallModel.Fz,
         para->getParD(level)->neighborX,
         para->getParD(level)->neighborY,
         para->getParD(level)->neighborZ,
         para->getParD(level)->numberOfNodes,
         para->getParD(level)->isEvenTimestep);
      getLastCudaError("QSlipDeviceComp27 execution failed");
}

//////////////////////////////////////////////////////////////////////////
void BBStressDev27(Parameter *para,  QforBoundaryConditions* boundaryCondition, const int level)
{
   dim3 grid = vf::cuda::getCudaGrid( para->getParD(level)->numberofthreads, boundaryCondition->numberOfBCnodes);
   dim3 threads(para->getParD(level)->numberofthreads, 1, 1 );

   BBStressDevice27<<< grid, threads >>> (
      para->getParD(level)->distributions.f[0],
      boundaryCondition->k,
      boundaryCondition->kN,
      boundaryCondition->q27[0],
      boundaryCondition->numberOfBCnodes,
      para->getParD(level)->velocityX,
      para->getParD(level)->velocityY,
      para->getParD(level)->velocityY,
      boundaryCondition->normalX,
      boundaryCondition->normalY,
      boundaryCondition->normalZ,
      boundaryCondition->Vx,
      boundaryCondition->Vy,
      boundaryCondition->Vz,
      boundaryCondition->Vx1,
      boundaryCondition->Vy1,
      boundaryCondition->Vz1,
      para->getParD(level)->wallModel.samplingOffset,
      para->getParD(level)->wallModel.z0,
      para->getHasWallModelMonitor(),
      para->getParD(level)->wallModel.u_star,
      para->getParD(level)->wallModel.Fx,
      para->getParD(level)->wallModel.Fy,
      para->getParD(level)->wallModel.Fz,
      para->getParD(level)->neighborX,
      para->getParD(level)->neighborY,
      para->getParD(level)->neighborZ,
      para->getParD(level)->numberOfNodes,
      para->getParD(level)->isEvenTimestep);
      getLastCudaError("BBStressDevice27 execution failed");
}

//////////////////////////////////////////////////////////////////////////
void BBStressPressureDev27(Parameter *para,  QforBoundaryConditions* boundaryCondition, const int level)
{
   dim3 grid = vf::cuda::getCudaGrid( para->getParD(level)->numberofthreads, boundaryCondition->numberOfBCnodes);
   dim3 threads(para->getParD(level)->numberofthreads, 1, 1 );

   BBStressPressureDevice27<<< grid, threads >>> (
      para->getParD(level)->distributions.f[0],
      boundaryCondition->k,
      boundaryCondition->kN,
      boundaryCondition->q27[0],
      boundaryCondition->numberOfBCnodes,
      para->getParD(level)->velocityX,
      para->getParD(level)->velocityY,
      para->getParD(level)->velocityY,
      boundaryCondition->normalX,
      boundaryCondition->normalY,
      boundaryCondition->normalZ,
      boundaryCondition->Vx,
      boundaryCondition->Vy,
      boundaryCondition->Vz,
      boundaryCondition->Vx1,
      boundaryCondition->Vy1,
      boundaryCondition->Vz1,
      para->getParD(level)->wallModel.samplingOffset,
      para->getParD(level)->wallModel.z0,
      para->getHasWallModelMonitor(),
      para->getParD(level)->wallModel.u_star,
      para->getParD(level)->wallModel.Fx,
      para->getParD(level)->wallModel.Fy,
      para->getParD(level)->wallModel.Fz,
      para->getParD(level)->neighborX,
      para->getParD(level)->neighborY,
      para->getParD(level)->neighborZ,
      para->getParD(level)->numberOfNodes,
      para->getParD(level)->isEvenTimestep);
      getLastCudaError("BBStressDevice27 execution failed");
}

//////////////////////////////////////////////////////////////////////////
void QPressDev27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QPressDevice27<<< grid, threads >>> (
      boundaryCondition->RhoBC,
      parameterDevice->distributions.f[0],
      boundaryCondition->k,
      boundaryCondition->q27[0],
      boundaryCondition->numberOfBCnodes,
      parameterDevice->omega,
      parameterDevice->neighborX,
      parameterDevice->neighborY,
      parameterDevice->neighborZ,
      parameterDevice->numberOfNodes,
      parameterDevice->isEvenTimestep);
   getLastCudaError("QPressDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevAntiBB27(  unsigned int numberOfThreads,
                                    real* rhoBC,
									real* vx,
									real* vy,
									real* vz,
									real* DD,
									int* k_Q,
									real* QQ,
									int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceAntiBB27<<< grid.grid, grid.threads >>>( rhoBC,
												vx,
												vy,
												vz,
												DD,
												k_Q,
												QQ,
												numberOfBCnodes,
												om1,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep);
   getLastCudaError("QPressDeviceAntiBB27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevFixBackflow27( unsigned int numberOfThreads,
                                        real* rhoBC,
                                        real* DD,
                                        int* k_Q,
                                        unsigned int numberOfBCnodes,
                                        real om1,
                                        unsigned int* neighborX,
                                        unsigned int* neighborY,
                                        unsigned int* neighborZ,
                                        unsigned int size_Mat,
                                        bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceFixBackflow27<<< grid.grid, grid.threads >>> (  rhoBC,
                                                         DD,
                                                         k_Q,
                                                         numberOfBCnodes,
                                                         om1,
                                                         neighborX,
                                                         neighborY,
                                                         neighborZ,
                                                         size_Mat,
                                                         isEvenTimestep);
   getLastCudaError("QPressDeviceFixBackflow27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevDirDepBot27(  unsigned int numberOfThreads,
                                       real* rhoBC,
                                       real* DD,
                                       int* k_Q,
                                       unsigned int numberOfBCnodes,
                                       real om1,
                                       unsigned int* neighborX,
                                       unsigned int* neighborY,
                                       unsigned int* neighborZ,
                                       unsigned int size_Mat,
                                       bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceDirDepBot27<<< grid.grid, grid.threads >>> ( rhoBC,
                                                      DD,
                                                      k_Q,
                                                      numberOfBCnodes,
                                                      om1,
                                                      neighborX,
                                                      neighborY,
                                                      neighborZ,
                                                      size_Mat,
                                                      isEvenTimestep);
   getLastCudaError("QPressDeviceDirDepBot27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressNoRhoDev27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QPressNoRhoDevice27<<< grid, threads >>> (
         boundaryCondition->RhoBC,
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->kN,
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep,
         vf::lbm::dir::DIR_P00);
   getLastCudaError("QPressNoRhoDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressZeroRhoOutflowDev27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QPressZeroRhoOutflowDevice27<<< grid, threads >>> (
         boundaryCondition->RhoBC,
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->kN,
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep,
         vf::lbm::dir::DIR_P00,
         parameterDevice->outflowPressureCorrectionFactor);
   getLastCudaError("QPressZeroRhoOutflowDev27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QInflowScaleByPressDev27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QInflowScaleByPressDevice27<<< grid, threads >>> (
           boundaryCondition->RhoBC,
           parameterDevice->distributions.f[0],
           boundaryCondition->k,
           boundaryCondition->kN,
           boundaryCondition->numberOfBCnodes,
           parameterDevice->omega,
           parameterDevice->neighborX,
           parameterDevice->neighborY,
           parameterDevice->neighborZ,
           parameterDevice->numberOfNodes,
           parameterDevice->isEvenTimestep);
   getLastCudaError("QInflowScaleByPressDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevOld27(  unsigned int numberOfThreads,
                                     real* rhoBC,
                                     real* DD,
                                     int* k_Q,
                                     int* k_N,
                                     unsigned int numberOfBCnodes,
                                     real om1,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceOld27<<< grid.grid, grid.threads >>> ( rhoBC,
                                                DD,
                                                k_Q,
                                                k_N,
                                                numberOfBCnodes,
                                                om1,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                isEvenTimestep);
   getLastCudaError("QPressDeviceOld27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevIncompNEQ27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QPressDeviceIncompNEQ27<<< grid, threads >>> (
         boundaryCondition->RhoBC,
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->kN,
         boundaryCondition->numberOfBCnodes,
         parameterDevice->omega,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("QPressDeviceIncompNEQ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevNEQ27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   QPressDeviceNEQ27<<< grid, threads >>> (
        boundaryCondition->RhoBC,
        parameterDevice->distributions.f[0],
        boundaryCondition->k,
        boundaryCondition->kN,
        boundaryCondition->numberOfBCnodes,
        parameterDevice->omega,
        parameterDevice->neighborX,
        parameterDevice->neighborY,
        parameterDevice->neighborZ,
        parameterDevice->numberOfNodes,
        parameterDevice->isEvenTimestep);
   getLastCudaError("QPressDevNEQ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevEQZ27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

      QPressDeviceEQZ27<<< grid, threads >>> (
            boundaryCondition->RhoBC,
            parameterDevice->distributions.f[0],
            boundaryCondition->k,
            boundaryCondition->kN,
            parameterDevice->kDistTestRE.f[0],
            boundaryCondition->numberOfBCnodes,
            parameterDevice->omega,
            parameterDevice->neighborX,
            parameterDevice->neighborY,
            parameterDevice->neighborZ,
            parameterDevice->numberOfNodes,
            parameterDevice->isEvenTimestep);
      getLastCudaError("QPressDeviceEQZ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevZero27(unsigned int numberOfThreads,
                                real* DD,
                                int* k_Q,
                                unsigned int numberOfBCnodes,
                                unsigned int* neighborX,
                                unsigned int* neighborY,
                                unsigned int* neighborZ,
                                unsigned int size_Mat,
                                bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceZero27<<< grid.grid, grid.threads >>> (DD,
                                                k_Q,
                                                numberOfBCnodes,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                isEvenTimestep);
   getLastCudaError("QPressDeviceOld27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDevFake27(     unsigned int numberOfThreads,
                                     real* rhoBC,
                                     real* DD,
                                     int* k_Q,
                                     int* k_N,
                                     unsigned int numberOfBCnodes,
                                     real om1,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      QPressDeviceFake27<<< grid.grid, grid.threads >>> (rhoBC,
                                                DD,
                                                k_Q,
                                                k_N,
                                                numberOfBCnodes,
                                                om1,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                isEvenTimestep);
      getLastCudaError("QPressDeviceFake27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void BBDev27(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
   dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
   dim3 threads(parameterDevice->numberofthreads, 1, 1 );

   BBDevice27<<< grid, threads >>> (
         parameterDevice->distributions.f[0],
         boundaryCondition->k,
         boundaryCondition->q27[0],
         boundaryCondition->numberOfBCnodes,
         parameterDevice->neighborX,
         parameterDevice->neighborY,
         parameterDevice->neighborZ,
         parameterDevice->numberOfNodes,
         parameterDevice->isEvenTimestep);
   getLastCudaError("BBDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPressDev27_IntBB(  unsigned int numberOfThreads,
									real* rho,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

	QPressDevice27_IntBB<<< grid.grid, grid.threads >>> (rho,
													DD,
													k_Q,
													QQ,
													numberOfBCnodes,
													om1,
													neighborX,
													neighborY,
													neighborZ,
													size_Mat,
													isEvenTimestep);
	getLastCudaError("QPressDevice27_IntBB execution failed");
}
// TODO: https://git.rz.tu-bs.de/irmb/VirtualFluids_dev/-/issues/29
//////////////////////////////////////////////////////////////////////////
void PressSchlaffer27(unsigned int numberOfThreads,
                                 real* rhoBC,
                                 real* DD,
                                 real* vx0,
                                 real* vy0,
                                 real* vz0,
                                 real* deltaVz0,
                                 int* k_Q,
                                 int* k_N,
                                 int numberOfBCnodes,
                                 real om1,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 unsigned int size_Mat,
                                 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   PressSchlaff27<<< grid.grid, grid.threads >>>(  rhoBC,
                                             DD,
                                             vx0,
                                             vy0,
                                             vz0,
                                             deltaVz0,
                                             k_Q,
                                             k_N,
                                             numberOfBCnodes,
                                             om1,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             isEvenTimestep);
   getLastCudaError("PressSchlaff27 execution failed");
}
// TODO: https://git.rz.tu-bs.de/irmb/VirtualFluids_dev/-/issues/29
//////////////////////////////////////////////////////////////////////////
void VelSchlaffer27(  unsigned int numberOfThreads,
                                 int t,
                                 real* DD,
                                 real* vz0,
                                 real* deltaVz0,
                                 int* k_Q,
                                 int* k_N,
                                 int numberOfBCnodes,
                                 real om1,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 unsigned int size_Mat,
                                 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   VelSchlaff27<<< grid.grid, grid.threads >>>( t,
                                          DD,
                                          vz0,
                                          deltaVz0,
                                          k_Q,
                                          k_N,
                                          numberOfBCnodes,
                                          om1,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          isEvenTimestep);
      getLastCudaError("VelSchlaff27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void QPrecursorDevCompZeroPress(  uint numberOfThreads, real tRatio,
                                             real* DD, real* QQ, int* k_Q, 
                                             uint sizeQ, uint numberOfBCnodes,
                                             real omega, real velocityRatio,
                                             uint* neighborX, uint* neighborY, uint* neighborZ,
                                             uint* neighborsNT, uint* neighborsNB, uint* neighborsST, uint* neighborsSB,
                                             real* weightsNT, real* weightsNB, real* weightsST, real* weightsSB,
                                             real* vxLast, real* vyLast, real* vzLast,
                                             real* vxCurrent, real* vyCurrent, real* vzCurrent,
                                             real velocityX, real velocityY, real velocityZ, 
                                             unsigned long long size_Mat, bool evenOrOdd)
{

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPrecursorDeviceCompZeroPress<<< grid.grid, grid.threads >>>(k_Q, numberOfBCnodes, sizeQ, omega, DD, QQ,
                                                               neighborX, neighborY, neighborZ,
                                                               neighborsNT, neighborsNB, neighborsST, neighborsSB,
                                                               weightsNT, weightsNB, weightsST, weightsSB,
                                                               vxLast, vyLast, vzLast,
                                                               vxCurrent, vyCurrent, vzCurrent, 
                                                               velocityX, velocityY, velocityZ, 
                                                               tRatio, velocityRatio, size_Mat, evenOrOdd);
   getLastCudaError("QPrecursorDeviceCompZeroPress execution failed"); 


}
//////////////////////////////////////////////////////////////////////////
extern "C" void PropVelo(   unsigned int numberOfThreads,
                            unsigned int* neighborX,
                            unsigned int* neighborY,
                            unsigned int* neighborZ,
                            real* rho,
                            real* ux,
                            real* uy,
                            real* uz,
                            int* k_Q,
							unsigned int size_Prop,
                            unsigned int size_Mat,
                            unsigned int* bcMatD,
                            real* DD,
                            bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Prop);

      PropellerBC<<< grid.grid, grid.threads >>>(neighborX,
                                       neighborY,
                                       neighborZ,
                                       rho,
                                       ux,
                                       uy,
                                       uz,
									   k_Q,
									   size_Prop,
                                       size_Mat,
									   bcMatD,
                                       DD,
                                       EvenOrOdd);
      getLastCudaError("PropellerBC execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF27( real* DC,
                        real* DF,
                        unsigned int* neighborCX,
                        unsigned int* neighborCY,
                        unsigned int* neighborCZ,
                        unsigned int* neighborFX,
                        unsigned int* neighborFY,
                        unsigned int* neighborFZ,
                        unsigned int size_MatC,
                        unsigned int size_MatF,
                        bool isEvenTimestep,
                        unsigned int* posCSWB,
                        unsigned int* posFSWB,
                        unsigned int kCF,
                        real omCoarse,
                        real omFine,
                        real nu,
                        unsigned int nxC,
                        unsigned int nyC,
                        unsigned int nxF,
                        unsigned int nyF,
                        unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);
   
      scaleCF27<<< grid.grid, grid.threads >>> ( DC,
                                             DF,
                                             neighborCX,
                                             neighborCY,
                                             neighborCZ,
                                             neighborFX,
                                             neighborFY,
                                             neighborFZ,
                                             size_MatC,
                                             size_MatF,
                                             isEvenTimestep,
                                             posCSWB,
                                             posFSWB,
                                             kCF,
                                             omCoarse,
                                             omFine,
                                             nu,
                                             nxC,
                                             nyC,
                                             nxF,
                                             nyF);
      getLastCudaError("scaleCF27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCFEff27(real* DC,
                             real* DF,
                             unsigned int* neighborCX,
                             unsigned int* neighborCY,
                             unsigned int* neighborCZ,
                             unsigned int* neighborFX,
                             unsigned int* neighborFY,
                             unsigned int* neighborFZ,
                             unsigned int size_MatC,
                             unsigned int size_MatF,
                             bool isEvenTimestep,
                             unsigned int* posCSWB,
                             unsigned int* posFSWB,
                             unsigned int kCF,
                             real omCoarse,
                             real omFine,
                             real nu,
                             unsigned int nxC,
                             unsigned int nyC,
                             unsigned int nxF,
                             unsigned int nyF,
                             unsigned int numberOfThreads,
                             OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFEff27<<< grid.grid, grid.threads >>> ( DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offCF);
      getLastCudaError("scaleCFEff27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCFLast27(real* DC,
                              real* DF,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posCSWB,
                              unsigned int* posFSWB,
                              unsigned int kCF,
                              real omCoarse,
                              real omFine,
                              real nu,
                              unsigned int nxC,
                              unsigned int nyC,
                              unsigned int nxF,
                              unsigned int nyF,
                              unsigned int numberOfThreads,
                              OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFLast27<<< grid.grid, grid.threads >>> (DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offCF);
      getLastCudaError("scaleCFLast27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCFpress27(  real* DC,
                                 real* DF,
                                 unsigned int* neighborCX,
                                 unsigned int* neighborCY,
                                 unsigned int* neighborCZ,
                                 unsigned int* neighborFX,
                                 unsigned int* neighborFY,
                                 unsigned int* neighborFZ,
                                 unsigned int size_MatC,
                                 unsigned int size_MatF,
                                 bool isEvenTimestep,
                                 unsigned int* posCSWB,
                                 unsigned int* posFSWB,
                                 unsigned int kCF,
                                 real omCoarse,
                                 real omFine,
                                 real nu,
                                 unsigned int nxC,
                                 unsigned int nyC,
                                 unsigned int nxF,
                                 unsigned int nyF,
                                 unsigned int numberOfThreads,
                                 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFpress27<<< grid.grid, grid.threads >>>(DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offCF);
      getLastCudaError("scaleCFpress27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_Fix_27(  real* DC,
                                 real* DF,
                                 unsigned int* neighborCX,
                                 unsigned int* neighborCY,
                                 unsigned int* neighborCZ,
                                 unsigned int* neighborFX,
                                 unsigned int* neighborFY,
                                 unsigned int* neighborFZ,
                                 unsigned int size_MatC,
                                 unsigned int size_MatF,
                                 bool isEvenTimestep,
                                 unsigned int* posCSWB,
                                 unsigned int* posFSWB,
                                 unsigned int kCF,
                                 real omCoarse,
                                 real omFine,
                                 real nu,
                                 unsigned int nxC,
                                 unsigned int nyC,
                                 unsigned int nxF,
                                 unsigned int nyF,
                                 unsigned int numberOfThreads,
                                 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_Fix_27<<< grid.grid, grid.threads >>>(DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offCF);
      getLastCudaError("scaleCF_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_Fix_comp_27( real* DC,
									 real* DF,
									 unsigned int* neighborCX,
									 unsigned int* neighborCY,
									 unsigned int* neighborCZ,
									 unsigned int* neighborFX,
									 unsigned int* neighborFY,
									 unsigned int* neighborFZ,
									 unsigned int size_MatC,
									 unsigned int size_MatF,
									 bool isEvenTimestep,
									 unsigned int* posCSWB,
									 unsigned int* posFSWB,
									 unsigned int kCF,
									 real omCoarse,
									 real omFine,
									 real nu,
									 unsigned int nxC,
									 unsigned int nyC,
									 unsigned int nxF,
									 unsigned int nyF,
									 unsigned int numberOfThreads,
									 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_Fix_comp_27<<< grid.grid, grid.threads >>>(   DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_0817_comp_27(real* DC,
									 real* DF,
									 unsigned int* neighborCX,
									 unsigned int* neighborCY,
									 unsigned int* neighborCZ,
									 unsigned int* neighborFX,
									 unsigned int* neighborFY,
									 unsigned int* neighborFZ,
									 unsigned int size_MatC,
									 unsigned int size_MatF,
									 bool isEvenTimestep,
									 unsigned int* posCSWB,
									 unsigned int* posFSWB,
									 unsigned int kCF,
									 real omCoarse,
									 real omFine,
									 real nu,
									 unsigned int nxC,
									 unsigned int nyC,
									 unsigned int nxF,
									 unsigned int nyF,
									 unsigned int numberOfThreads,
									 OffCF offCF,
                            ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_0817_comp_27<<< grid.grid, grid.threads, 0, stream >>>(  DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_0817_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_comp_D3Q27F3_2018(real* DC,
										  real* DF,
										  real* G6,
										  unsigned int* neighborCX,
										  unsigned int* neighborCY,
										  unsigned int* neighborCZ,
										  unsigned int* neighborFX,
										  unsigned int* neighborFY,
										  unsigned int* neighborFZ,
										  unsigned int size_MatC,
										  unsigned int size_MatF,
										  bool isEvenTimestep,
										  unsigned int* posCSWB,
										  unsigned int* posFSWB,
										  unsigned int kCF,
										  real omCoarse,
										  real omFine,
										  real nu,
										  unsigned int nxC,
										  unsigned int nyC,
										  unsigned int nxF,
										  unsigned int nyF,
										  unsigned int numberOfThreads,
										  OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_comp_D3Q27F3_2018 <<< grid.grid, grid.threads >>>(DC,
															DF,
															G6,
															neighborCX,
															neighborCY,
															neighborCZ,
															neighborFX,
															neighborFY,
															neighborFZ,
															size_MatC,
															size_MatF,
															isEvenTimestep,
															posCSWB,
															posFSWB,
															kCF,
															omCoarse,
															omFine,
															nu,
															nxC,
															nyC,
															nxF,
															nyF,
															offCF);
      getLastCudaError("scaleCF_comp_D3Q27F3_2018 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_comp_D3Q27F3(real* DC,
									 real* DF,
									 real* G6,
									 unsigned int* neighborCX,
									 unsigned int* neighborCY,
									 unsigned int* neighborCZ,
									 unsigned int* neighborFX,
									 unsigned int* neighborFY,
									 unsigned int* neighborFZ,
									 unsigned int size_MatC,
									 unsigned int size_MatF,
									 bool isEvenTimestep,
									 unsigned int* posCSWB,
									 unsigned int* posFSWB,
									 unsigned int kCF,
									 real omCoarse,
									 real omFine,
									 real nu,
									 unsigned int nxC,
									 unsigned int nyC,
									 unsigned int nxF,
									 unsigned int nyF,
									 unsigned int numberOfThreads,
									 OffCF offCF,
                            ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_comp_D3Q27F3 <<< grid.grid, grid.threads, 0, stream >>>( DC,
														DF,
														G6,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_comp_D3Q27F3 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_staggered_time_comp_27(  real* DC,
												 real* DF,
												 unsigned int* neighborCX,
												 unsigned int* neighborCY,
												 unsigned int* neighborCZ,
												 unsigned int* neighborFX,
												 unsigned int* neighborFY,
												 unsigned int* neighborFZ,
												 unsigned int size_MatC,
												 unsigned int size_MatF,
												 bool isEvenTimestep,
												 unsigned int* posCSWB,
												 unsigned int* posFSWB,
												 unsigned int kCF,
												 real omCoarse,
												 real omFine,
												 real nu,
												 unsigned int nxC,
												 unsigned int nyC,
												 unsigned int nxF,
												 unsigned int nyF,
												 unsigned int numberOfThreads,
												 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_staggered_time_comp_27<<< grid.grid, grid.threads >>>(    DC,
																	DF,
																	neighborCX,
																	neighborCY,
																	neighborCZ,
																	neighborFX,
																	neighborFY,
																	neighborFZ,
																	size_MatC,
																	size_MatF,
																	isEvenTimestep,
																	posCSWB,
																	posFSWB,
																	kCF,
																	omCoarse,
																	omFine,
																	nu,
																	nxC,
																	nyC,
																	nxF,
																	nyF,
																	offCF);
      getLastCudaError("scaleCF_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_RhoSq_comp_27(   real* DC,
										 real* DF,
										 unsigned int* neighborCX,
										 unsigned int* neighborCY,
										 unsigned int* neighborCZ,
										 unsigned int* neighborFX,
										 unsigned int* neighborFY,
										 unsigned int* neighborFZ,
										 unsigned int size_MatC,
										 unsigned int size_MatF,
										 bool isEvenTimestep,
										 unsigned int* posCSWB,
										 unsigned int* posFSWB,
										 unsigned int kCF,
										 real omCoarse,
										 real omFine,
										 real nu,
										 unsigned int nxC,
										 unsigned int nyC,
										 unsigned int nxF,
										 unsigned int nyF,
										 unsigned int numberOfThreads,
										 OffCF offCF,
                               ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_RhoSq_comp_27<<< grid.grid, grid.threads, 0, stream >>>( DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_RhoSq_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_RhoSq_3rdMom_comp_27(real* DC,
											 real* DF,
											 unsigned int* neighborCX,
											 unsigned int* neighborCY,
											 unsigned int* neighborCZ,
											 unsigned int* neighborFX,
											 unsigned int* neighborFY,
											 unsigned int* neighborFZ,
											 unsigned int size_MatC,
											 unsigned int size_MatF,
											 bool isEvenTimestep,
											 unsigned int* posCSWB,
											 unsigned int* posFSWB,
											 unsigned int kCF,
											 real omCoarse,
											 real omFine,
											 real nu,
											 unsigned int nxC,
											 unsigned int nyC,
											 unsigned int nxF,
											 unsigned int nyF,
											 unsigned int numberOfThreads,
											 OffCF offCF,
                                  ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_RhoSq_3rdMom_comp_27<<< grid.grid, grid.threads, 0, stream >>>(  DC,
																DF,
																neighborCX,
																neighborCY,
																neighborCZ,
																neighborFX,
																neighborFY,
																neighborFZ,
																size_MatC,
																size_MatF,
																isEvenTimestep,
																posCSWB,
																posFSWB,
																kCF,
																omCoarse,
																omFine,
																nu,
																nxC,
																nyC,
																nxF,
																nyF,
																offCF);
      getLastCudaError("scaleCF_RhoSq_3rdMom_comp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_AA2016_comp_27(real* DC,
									   real* DF,
									   unsigned int* neighborCX,
									   unsigned int* neighborCY,
									   unsigned int* neighborCZ,
									   unsigned int* neighborFX,
									   unsigned int* neighborFY,
									   unsigned int* neighborFZ,
									   unsigned int size_MatC,
									   unsigned int size_MatF,
									   bool isEvenTimestep,
									   unsigned int* posCSWB,
									   unsigned int* posFSWB,
									   unsigned int kCF,
									   real omCoarse,
									   real omFine,
									   real nu,
									   unsigned int nxC,
									   unsigned int nyC,
									   unsigned int nxF,
									   unsigned int nyF,
									   unsigned int numberOfThreads,
									   OffCF offCF,
                              ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_AA2016_comp_27<<< grid.grid, grid.threads, 0, stream >>>(DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_AA2016_comp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCF_NSPress_27(  real* DC,
									 real* DF,
									 unsigned int* neighborCX,
									 unsigned int* neighborCY,
									 unsigned int* neighborCZ,
									 unsigned int* neighborFX,
									 unsigned int* neighborFY,
									 unsigned int* neighborFZ,
									 unsigned int size_MatC,
									 unsigned int size_MatF,
									 bool isEvenTimestep,
									 unsigned int* posCSWB,
									 unsigned int* posFSWB,
									 unsigned int kCF,
									 real omCoarse,
									 real omFine,
									 real nu,
									 unsigned int nxC,
									 unsigned int nyC,
									 unsigned int nxF,
									 unsigned int nyF,
									 unsigned int numberOfThreads,
									 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_NSPress_27<<< grid.grid, grid.threads >>>(DC,
													DF,
													neighborCX,
													neighborCY,
													neighborCZ,
													neighborFX,
													neighborFY,
													neighborFZ,
													size_MatC,
													size_MatF,
													isEvenTimestep,
													posCSWB,
													posFSWB,
													kCF,
													omCoarse,
													omFine,
													nu,
													nxC,
													nyC,
													nxF,
													nyF,
													offCF);
      getLastCudaError("scaleCF_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCFThSMG7(   real* DC,
                                 real* DF,
                                 real* DD7C,
                                 real* DD7F,
                                 unsigned int* neighborCX,
                                 unsigned int* neighborCY,
                                 unsigned int* neighborCZ,
                                 unsigned int* neighborFX,
                                 unsigned int* neighborFY,
                                 unsigned int* neighborFZ,
                                 unsigned int size_MatC,
                                 unsigned int size_MatF,
                                 bool isEvenTimestep,
                                 unsigned int* posCSWB,
                                 unsigned int* posFSWB,
                                 unsigned int kCF,
                                 real nu,
                                 real diffusivity_fine,
                                 unsigned int numberOfThreads,
                                 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFThSMG7<<< grid.grid, grid.threads >>> (DC,
                                                DF,
                                                DD7C,
                                                DD7F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                nu,
                                                diffusivity_fine,
                                                offCF);
      getLastCudaError("scaleCFThSMG7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCFThS7(  real* DC,
                              real* DF,
                              real* DD7C,
                              real* DD7F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posCSWB,
                              unsigned int* posFSWB,
                              unsigned int kCF,
                              real nu,
                              real diffusivity_fine,
                              unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFThS7<<< grid.grid, grid.threads >>> (  DC,
                                                DF,
                                                DD7C,
                                                DD7F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                nu,
                                                diffusivity_fine);
      getLastCudaError("scaleCFThS7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleCFThS27( real* DC,
                              real* DF,
                              real* DD27C,
                              real* DD27F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posCSWB,
                              unsigned int* posFSWB,
                              unsigned int kCF,
                              real nu,
                              real diffusivity_fine,
                              unsigned int numberOfThreads,
							  OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFThS27<<< grid.grid, grid.threads >>> ( DC,
                                                DF,
                                                DD27C,
                                                DD27F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                nu,
                                                diffusivity_fine,
										        offCF);
      getLastCudaError("scaleCFThS27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC27( real* DC,
                           real* DF,
                           unsigned int* neighborCX,
                           unsigned int* neighborCY,
                           unsigned int* neighborCZ,
                           unsigned int* neighborFX,
                           unsigned int* neighborFY,
                           unsigned int* neighborFZ,
                           unsigned int size_MatC,
                           unsigned int size_MatF,
                           bool isEvenTimestep,
                           unsigned int* posC,
                           unsigned int* posFSWB,
                           unsigned int kFC,
                           real omCoarse,
                           real omFine,
                           real nu,
                           unsigned int nxC,
                           unsigned int nyC,
                           unsigned int nxF,
                           unsigned int nyF,
                           unsigned int numberOfThreads)
{
   
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC27<<< grid.grid, grid.threads >>> ( DC,
                                             DF,
                                             neighborCX,
                                             neighborCY,
                                             neighborCZ,
                                             neighborFX,
                                             neighborFY,
                                             neighborFZ,
                                             size_MatC,
                                             size_MatF,
                                             isEvenTimestep,
                                             posC,
                                             posFSWB,
                                             kFC,
                                             omCoarse,
                                             omFine,
                                             nu,
                                             nxC,
                                             nyC,
                                             nxF,
                                             nyF);
      getLastCudaError("scaleFC27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFCEff27(real* DC,
                             real* DF,
                             unsigned int* neighborCX,
                             unsigned int* neighborCY,
                             unsigned int* neighborCZ,
                             unsigned int* neighborFX,
                             unsigned int* neighborFY,
                             unsigned int* neighborFZ,
                             unsigned int size_MatC,
                             unsigned int size_MatF,
                             bool isEvenTimestep,
                             unsigned int* posC,
                             unsigned int* posFSWB,
                             unsigned int kFC,
                             real omCoarse,
                             real omFine,
                             real nu,
                             unsigned int nxC,
                             unsigned int nyC,
                             unsigned int nxF,
                             unsigned int nyF,
                             unsigned int numberOfThreads,
                             OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCEff27<<< grid.grid, grid.threads >>> ( DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posC,
                                                posFSWB,
                                                kFC,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offFC);
      getLastCudaError("scaleFCEff27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFCLast27(real* DC,
                              real* DF,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real omCoarse,
                              real omFine,
                              real nu,
                              unsigned int nxC,
                              unsigned int nyC,
                              unsigned int nxF,
                              unsigned int nyF,
                              unsigned int numberOfThreads,
                              OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCLast27<<< grid.grid, grid.threads >>> (DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posC,
                                                posFSWB,
                                                kFC,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offFC);
      getLastCudaError("Kernel execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFCpress27(real* DC,
                              real* DF,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real omCoarse,
                              real omFine,
                              real nu,
                              unsigned int nxC,
                              unsigned int nyC,
                              unsigned int nxF,
                              unsigned int nyF,
                              unsigned int numberOfThreads,
                              OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCpress27<<< grid.grid, grid.threads >>> (  DC,
                                                   DF,
                                                   neighborCX,
                                                   neighborCY,
                                                   neighborCZ,
                                                   neighborFX,
                                                   neighborFY,
                                                   neighborFZ,
                                                   size_MatC,
                                                   size_MatF,
                                                   isEvenTimestep,
                                                   posC,
                                                   posFSWB,
                                                   kFC,
                                                   omCoarse,
                                                   omFine,
                                                   nu,
                                                   nxC,
                                                   nyC,
                                                   nxF,
                                                   nyF,
                                                   offFC);
      getLastCudaError("scaleFCpress27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_Fix_27(real* DC,
                              real* DF,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real omCoarse,
                              real omFine,
                              real nu,
                              unsigned int nxC,
                              unsigned int nyC,
                              unsigned int nxF,
                              unsigned int nyF,
                              unsigned int numberOfThreads,
                              OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_Fix_27<<< grid.grid, grid.threads >>> (  DC,
                                                   DF,
                                                   neighborCX,
                                                   neighborCY,
                                                   neighborCZ,
                                                   neighborFX,
                                                   neighborFY,
                                                   neighborFZ,
                                                   size_MatC,
                                                   size_MatF,
                                                   isEvenTimestep,
                                                   posC,
                                                   posFSWB,
                                                   kFC,
                                                   omCoarse,
                                                   omFine,
                                                   nu,
                                                   nxC,
                                                   nyC,
                                                   nxF,
                                                   nyF,
                                                   offFC);
      getLastCudaError("scaleFC_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_Fix_comp_27(  real* DC,
									  real* DF,
									  unsigned int* neighborCX,
									  unsigned int* neighborCY,
									  unsigned int* neighborCZ,
									  unsigned int* neighborFX,
									  unsigned int* neighborFY,
									  unsigned int* neighborFZ,
									  unsigned int size_MatC,
									  unsigned int size_MatF,
									  bool isEvenTimestep,
									  unsigned int* posC,
									  unsigned int* posFSWB,
									  unsigned int kFC,
									  real omCoarse,
									  real omFine,
									  real nu,
									  unsigned int nxC,
									  unsigned int nyC,
									  unsigned int nxF,
									  unsigned int nyF,
									  unsigned int numberOfThreads,
									  OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_Fix_comp_27<<< grid.grid, grid.threads >>> ( DC,
													   DF,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_0817_comp_27( real* DC,
									  real* DF,
									  unsigned int* neighborCX,
									  unsigned int* neighborCY,
									  unsigned int* neighborCZ,
									  unsigned int* neighborFX,
									  unsigned int* neighborFY,
									  unsigned int* neighborFZ,
									  unsigned int size_MatC,
									  unsigned int size_MatF,
									  bool isEvenTimestep,
									  unsigned int* posC,
									  unsigned int* posFSWB,
									  unsigned int kFC,
									  real omCoarse,
									  real omFine,
									  real nu,
									  unsigned int nxC,
									  unsigned int nyC,
									  unsigned int nxF,
									  unsigned int nyF,
									  unsigned int numberOfThreads,
									  OffFC offFC,
                             ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_0817_comp_27<<< grid.grid, grid.threads, 0, stream >>> (DC,
													   DF,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_0817_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_comp_D3Q27F3_2018( real* DC,
										   real* DF,
										   real* G6,
										   unsigned int* neighborCX,
										   unsigned int* neighborCY,
										   unsigned int* neighborCZ,
										   unsigned int* neighborFX,
										   unsigned int* neighborFY,
										   unsigned int* neighborFZ,
										   unsigned int size_MatC,
										   unsigned int size_MatF,
										   bool isEvenTimestep,
										   unsigned int* posC,
										   unsigned int* posFSWB,
										   unsigned int kFC,
										   real omCoarse,
										   real omFine,
										   real nu,
										   unsigned int nxC,
										   unsigned int nyC,
										   unsigned int nxF,
										   unsigned int nyF,
										   unsigned int numberOfThreads,
										   OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

     scaleFC_comp_D3Q27F3_2018 <<< grid.grid, grid.threads >>> (DC,
															DF,
															G6,
															neighborCX,
															neighborCY,
															neighborCZ,
															neighborFX,
															neighborFY,
															neighborFZ,
															size_MatC,
															size_MatF,
															isEvenTimestep,
															posC,
															posFSWB,
															kFC,
															omCoarse,
															omFine,
															nu,
															nxC,
															nyC,
															nxF,
															nyF,
															offFC);
      getLastCudaError("scaleFC_comp_D3Q27F3_2018 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_comp_D3Q27F3( real* DC,
									  real* DF,
									  real* G6,
									  unsigned int* neighborCX,
									  unsigned int* neighborCY,
									  unsigned int* neighborCZ,
									  unsigned int* neighborFX,
									  unsigned int* neighborFY,
									  unsigned int* neighborFZ,
									  unsigned int size_MatC,
									  unsigned int size_MatF,
									  bool isEvenTimestep,
									  unsigned int* posC,
									  unsigned int* posFSWB,
									  unsigned int kFC,
									  real omCoarse,
									  real omFine,
									  real nu,
									  unsigned int nxC,
									  unsigned int nyC,
									  unsigned int nxF,
									  unsigned int nyF,
									  unsigned int numberOfThreads,
									  OffFC offFC,
                             ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

     scaleFC_comp_D3Q27F3 <<< grid.grid, grid.threads, 0, stream >>> (DC,
													   DF,
													   G6,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_0817_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_staggered_time_comp_27(   real* DC,
												  real* DF,
												  unsigned int* neighborCX,
												  unsigned int* neighborCY,
												  unsigned int* neighborCZ,
												  unsigned int* neighborFX,
												  unsigned int* neighborFY,
												  unsigned int* neighborFZ,
												  unsigned int size_MatC,
												  unsigned int size_MatF,
												  bool isEvenTimestep,
												  unsigned int* posC,
												  unsigned int* posFSWB,
												  unsigned int kFC,
												  real omCoarse,
												  real omFine,
												  real nu,
												  unsigned int nxC,
												  unsigned int nyC,
												  unsigned int nxF,
												  unsigned int nyF,
												  unsigned int numberOfThreads,
												  OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_staggered_time_comp_27<<< grid.grid, grid.threads >>> (  DC,
																   DF,
																   neighborCX,
																   neighborCY,
																   neighborCZ,
																   neighborFX,
																   neighborFY,
																   neighborFZ,
																   size_MatC,
																   size_MatF,
																   isEvenTimestep,
																   posC,
																   posFSWB,
																   kFC,
																   omCoarse,
																   omFine,
																   nu,
																   nxC,
																   nyC,
																   nxF,
																   nyF,
																   offFC);
      getLastCudaError("scaleFC_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_RhoSq_comp_27(real* DC,
									  real* DF,
									  unsigned int* neighborCX,
									  unsigned int* neighborCY,
									  unsigned int* neighborCZ,
									  unsigned int* neighborFX,
									  unsigned int* neighborFY,
									  unsigned int* neighborFZ,
									  unsigned int size_MatC,
									  unsigned int size_MatF,
									  bool isEvenTimestep,
									  unsigned int* posC,
									  unsigned int* posFSWB,
									  unsigned int kFC,
									  real omCoarse,
									  real omFine,
									  real nu,
									  unsigned int nxC,
									  unsigned int nyC,
									  unsigned int nxF,
									  unsigned int nyF,
									  unsigned int numberOfThreads,
									  OffFC offFC,
                             ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_RhoSq_comp_27<<<grid.grid, grid.threads, 0, stream>>>(
													   DC,
													   DF,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_RhoSq_27 execution failed");
}

//////////////////////////////////////////////////////////////////////////
void ScaleFC_RhoSq_3rdMom_comp_27( real* DC,
											  real* DF,
											  unsigned int* neighborCX,
											  unsigned int* neighborCY,
											  unsigned int* neighborCZ,
											  unsigned int* neighborFX,
											  unsigned int* neighborFY,
											  unsigned int* neighborFZ,
											  unsigned int size_MatC,
											  unsigned int size_MatF,
											  bool isEvenTimestep,
											  unsigned int* posC,
											  unsigned int* posFSWB,
											  unsigned int kFC,
											  real omCoarse,
											  real omFine,
											  real nu,
											  unsigned int nxC,
											  unsigned int nyC,
											  unsigned int nxF,
											  unsigned int nyF,
											  unsigned int numberOfThreads,
											  OffFC offFC,
                                   ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_RhoSq_3rdMom_comp_27<<< grid.grid, grid.threads, 0, stream >>>(DC,
															  DF,
															  neighborCX,
															  neighborCY,
															  neighborCZ,
															  neighborFX,
															  neighborFY,
															  neighborFZ,
															  size_MatC,
															  size_MatF,
															  isEvenTimestep,
															  posC,
															  posFSWB,
															  kFC,
															  omCoarse,
															  omFine,
															  nu,
															  nxC,
															  nyC,
															  nxF,
															  nyF,
															  offFC);
      getLastCudaError("scaleFC_RhoSq_3rdMom_comp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_AA2016_comp_27( real* DC,
										real* DF,
										unsigned int* neighborCX,
										unsigned int* neighborCY,
										unsigned int* neighborCZ,
										unsigned int* neighborFX,
										unsigned int* neighborFY,
										unsigned int* neighborFZ,
										unsigned int size_MatC,
										unsigned int size_MatF,
										bool isEvenTimestep,
										unsigned int* posC,
										unsigned int* posFSWB,
										unsigned int kFC,
										real omCoarse,
										real omFine,
										real nu,
										unsigned int nxC,
										unsigned int nyC,
										unsigned int nxF,
										unsigned int nyF,
										unsigned int numberOfThreads,
										OffFC offFC,
                              ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_AA2016_comp_27<<< grid.grid, grid.threads, 0, stream >>>(DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posC,
														posFSWB,
														kFC,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offFC);
      getLastCudaError("scaleFC_AA2016_comp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFC_NSPress_27(real* DC,
								  real* DF,
								  unsigned int* neighborCX,
								  unsigned int* neighborCY,
								  unsigned int* neighborCZ,
								  unsigned int* neighborFX,
								  unsigned int* neighborFY,
								  unsigned int* neighborFZ,
								  unsigned int size_MatC,
								  unsigned int size_MatF,
								  bool isEvenTimestep,
								  unsigned int* posC,
								  unsigned int* posFSWB,
								  unsigned int kFC,
								  real omCoarse,
								  real omFine,
								  real nu,
								  unsigned int nxC,
								  unsigned int nyC,
								  unsigned int nxF,
								  unsigned int nyF,
								  unsigned int numberOfThreads,
								  OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_NSPress_27<<< grid.grid, grid.threads >>> (  DC,
													   DF,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFCThSMG7(real* DC,
                              real* DF,
                              real* DD7C,
                              real* DD7F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real nu,
                              real diffusivity_coarse,
                              unsigned int numberOfThreads,
                              OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCThSMG7<<< grid.grid, grid.threads >>>( DC,
                                                DF,
                                                DD7C,
                                                DD7F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posC,
                                                posFSWB,
                                                kFC,
                                                nu,
                                                diffusivity_coarse,
                                                offFC);
      getLastCudaError("scaleFCThSMG7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFCThS7(  real* DC,
                              real* DF,
                              real* DD7C,
                              real* DD7F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real nu,
                              real diffusivity_coarse,
                              unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCThS7<<< grid.grid, grid.threads >>>(DC,
                                             DF,
                                             DD7C,
                                             DD7F,
                                             neighborCX,
                                             neighborCY,
                                             neighborCZ,
                                             neighborFX,
                                             neighborFY,
                                             neighborFZ,
                                             size_MatC,
                                             size_MatF,
                                             isEvenTimestep,
                                             posC,
                                             posFSWB,
                                             kFC,
                                             nu,
                                             diffusivity_coarse);
      getLastCudaError("scaleFCThS7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void ScaleFCThS27( real* DC,
                              real* DF,
                              real* DD27C,
                              real* DD27F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real nu,
                              real diffusivity_coarse,
                              unsigned int numberOfThreads,
							  OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCThS27<<< grid.grid, grid.threads >>>(  DC,
                                                DF,
                                                DD27C,
                                                DD27F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posC,
                                                posFSWB,
                                                kFC,
                                                nu,
                                                diffusivity_coarse,
												offFC);
      getLastCudaError("scaleFCThS27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void DragLiftPostD27(real* DD,
								int* k_Q,
								real* QQ,
								int numberOfBCnodes,
								double *DragX,
								double *DragY,
								double *DragZ,
								unsigned int* neighborX,
								unsigned int* neighborY,
								unsigned int* neighborZ,
								unsigned int size_Mat,
								bool isEvenTimestep,
								unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

	DragLiftPost27<<< grid.grid, grid.threads >>>(DD,
										k_Q,
										QQ,
										numberOfBCnodes,
										DragX,
										DragY,
										DragZ,
										neighborX,
										neighborY,
										neighborZ,
										size_Mat,
										isEvenTimestep);
	getLastCudaError("DragLift27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void DragLiftPreD27( real* DD,
								int* k_Q,
								real* QQ,
								int numberOfBCnodes,
								double *DragX,
								double *DragY,
								double *DragZ,
								unsigned int* neighborX,
								unsigned int* neighborY,
								unsigned int* neighborZ,
								unsigned int size_Mat,
								bool isEvenTimestep,
								unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

	DragLiftPre27<<< grid.grid, grid.threads >>>( DD,
										k_Q,
										QQ,
										numberOfBCnodes,
										DragX,
										DragY,
										DragZ,
										neighborX,
										neighborY,
										neighborZ,
										size_Mat,
										isEvenTimestep);
	getLastCudaError("DragLift27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcCPtop27(real* DD,
							int* cpIndex,
							int nonCp,
							double *cpPress,
							unsigned int* neighborX,
							unsigned int* neighborY,
							unsigned int* neighborZ,
							unsigned int size_Mat,
							bool isEvenTimestep,
							unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, nonCp);

	CalcCP27<<< grid.grid, grid.threads >>>(DD,
								  cpIndex,
								  nonCp,
								  cpPress,
								  neighborX,
								  neighborY,
								  neighborZ,
								  size_Mat,
								  isEvenTimestep);
	getLastCudaError("CalcCP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcCPbottom27( real* DD,
								int* cpIndex,
								int nonCp,
								double *cpPress,
								unsigned int* neighborX,
								unsigned int* neighborY,
								unsigned int* neighborZ,
								unsigned int size_Mat,
								bool isEvenTimestep,
								unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, nonCp);

	CalcCP27<<< grid.grid, grid.threads >>>(DD,
								  cpIndex,
								  nonCp,
								  cpPress,
								  neighborX,
								  neighborY,
								  neighborZ,
								  size_Mat,
								  isEvenTimestep);
	getLastCudaError("CalcCP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void GetSendFsPreDev27(real* DD,
								  real* bufferFs,
								  int* sendIndex,
								  int buffmax,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  unsigned int size_Mat,
								  bool isEvenTimestep,
								  unsigned int numberOfThreads,
								  hipStream_t stream)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	getSendFsPre27<<< grid.grid, grid.threads, 0, stream >>>(DD,
										bufferFs,
										sendIndex,
										buffmax,
										neighborX,
										neighborY,
										neighborZ,
										size_Mat,
										isEvenTimestep);
	getLastCudaError("getSendFsPre27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void GetSendFsPostDev27(real* DD,
								   real* bufferFs,
								   int* sendIndex,
								   int buffmax,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep,
								   unsigned int numberOfThreads,
								   hipStream_t stream)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	getSendFsPost27<<< grid.grid, grid.threads, 0, stream >>>(DD,
										 bufferFs,
										 sendIndex,
										 buffmax,
										 neighborX,
										 neighborY,
										 neighborZ,
										 size_Mat,
										 isEvenTimestep);
	getLastCudaError("getSendFsPost27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void SetRecvFsPreDev27(real* DD,
								  real* bufferFs,
								  int* recvIndex,
								  int buffmax,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  unsigned int size_Mat,
								  bool isEvenTimestep,
								  unsigned int numberOfThreads,
	                              hipStream_t stream)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	setRecvFsPre27<<< grid.grid, grid.threads, 0, stream >>>(DD,
										bufferFs,
										recvIndex,
										buffmax,
										neighborX,
										neighborY,
										neighborZ,
										size_Mat,
										isEvenTimestep);
	getLastCudaError("setRecvFsPre27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void SetRecvFsPostDev27(real* DD,
								   real* bufferFs,
								   int* recvIndex,
								   int buffmax,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep,
	                               unsigned int numberOfThreads,
	                               hipStream_t stream)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	setRecvFsPost27<<< grid.grid, grid.threads, 0, stream >>>(DD,
										 bufferFs,
										 recvIndex,
										 buffmax,
										 neighborX,
										 neighborY,
										 neighborZ,
										 size_Mat,
										 isEvenTimestep);
	getLastCudaError("setRecvFsPost27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void getSendGsDevF3(
	real* G6,
	real* bufferGs,
	int* sendIndex,
	int buffmax,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	bool isEvenTimestep,
	unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	getSendGsF3 <<< grid.grid, grid.threads >>> (
		G6,
		bufferGs,
		sendIndex,
		buffmax,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("getSendGsF3 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void setRecvGsDevF3(
	real* G6,
	real* bufferGs,
	int* recvIndex,
	int buffmax,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	bool isEvenTimestep,
	unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	setRecvGsF3 <<< grid.grid, grid.threads >>> (
		G6,
		bufferGs,
		recvIndex,
		buffmax,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("setRecvGsF3 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void WallFuncDev27(unsigned int numberOfThreads,
							  real* vx,
							  real* vy,
							  real* vz,
							  real* DD,
							  int* k_Q,
							  real* QQ,
							  unsigned int numberOfBCnodes,
							  real om1,
							  unsigned int* neighborX,
							  unsigned int* neighborY,
							  unsigned int* neighborZ,
							  unsigned int size_Mat,
							  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

      WallFunction27<<< grid.grid, grid.threads >>> (
											  vx,
											  vy,
											  vz,
											  DD,
											  k_Q,
											  QQ,
											  numberOfBCnodes,
											  om1,
											  neighborX,
											  neighborY,
											  neighborZ,
											  size_Mat,
											  isEvenTimestep);
      getLastCudaError("WallFunction27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void SetOutputWallVelocitySP27(unsigned int numberOfThreads,
										  real* vxD,
										  real* vyD,
										  real* vzD,
										  real* vxWall,
										  real* vyWall,
										  real* vzWall,
										  int numberOfWallNodes,
										  int* kWallNodes,
										  real* rhoD,
										  real* pressD,
										  unsigned int* geoD,
										  unsigned int* neighborX,
										  unsigned int* neighborY,
										  unsigned int* neighborZ,
										  unsigned int size_Mat,
										  real* DD,
										  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfWallNodes);

      LBSetOutputWallVelocitySP27<<< grid.grid, grid.threads >>> (	vxD,
															vyD,
															vzD,
															vxWall,
															vyWall,
															vzWall,
															numberOfWallNodes,
															kWallNodes,
															rhoD,
															pressD,
															geoD,
															neighborX,
															neighborY,
															neighborZ,
															size_Mat,
															DD,
															isEvenTimestep);
      getLastCudaError("LBSetOutputWallVelocitySP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void GetVelotoForce27(unsigned int numberOfThreads,
								 real* DD,
								 int* bcIndex,
								 int nonAtBC,
								 real* Vx,
								 real* Vy,
								 real* Vz,
								 unsigned int* neighborX,
								 unsigned int* neighborY,
								 unsigned int* neighborZ,
								 unsigned int size_Mat,
								 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, nonAtBC);

      GetVeloforForcing27<<< grid.grid, grid.threads >>> (DD,
												bcIndex,
												nonAtBC,
												Vx,
												Vy,
												Vz,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep);
      getLastCudaError("GetVeloforForcing27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void InitParticlesDevice(real* coordX,
									real* coordY,
									real* coordZ,
									real* coordParticleXlocal,
									real* coordParticleYlocal,
									real* coordParticleZlocal,
									real* coordParticleXglobal,
									real* coordParticleYglobal,
									real* coordParticleZglobal,
									real* veloParticleX,
									real* veloParticleY,
									real* veloParticleZ,
									real* randArray,
									unsigned int* particleID,
									unsigned int* cellBaseID,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int* neighborWSB,
									int level,
									unsigned int numberOfParticles,
									unsigned int size_Mat,
									unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfParticles);

   InitParticles<<< grid.grid, grid.threads >>> (coordX,
										coordY,
										coordZ,
										coordParticleXlocal,
										coordParticleYlocal,
										coordParticleZlocal,
										coordParticleXglobal,
										coordParticleYglobal,
										coordParticleZglobal,
										veloParticleX,
										veloParticleY,
										veloParticleZ,
										randArray,
										particleID,
										cellBaseID,
										bcMatD,
										neighborX,
										neighborY,
										neighborZ,
										neighborWSB,
										level,
										numberOfParticles,
										size_Mat);
      getLastCudaError("InitParticles execution failed");
}
//////////////////////////////////////////////////////////////////////////
void MoveParticlesDevice(real* coordX,
									real* coordY,
									real* coordZ,
									real* coordParticleXlocal,
									real* coordParticleYlocal,
									real* coordParticleZlocal,
									real* coordParticleXglobal,
									real* coordParticleYglobal,
									real* coordParticleZglobal,
									real* veloParticleX,
									real* veloParticleY,
									real* veloParticleZ,
									real* DD,
									real  omega,
									unsigned int* particleID,
									unsigned int* cellBaseID,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int* neighborWSB,
							        int level,
									unsigned int timestep,
									unsigned int numberOfTimesteps,
									unsigned int numberOfParticles,
									unsigned int size_Mat,
									unsigned int numberOfThreads,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfParticles);

   MoveParticles<<< grid.grid, grid.threads >>> (coordX,
										coordY,
										coordZ,
										coordParticleXlocal,
										coordParticleYlocal,
										coordParticleZlocal,
										coordParticleXglobal,
										coordParticleYglobal,
										coordParticleZglobal,
										veloParticleX,
										veloParticleY,
										veloParticleZ,
										DD,
										omega,
										particleID,
										cellBaseID,
										bcMatD,
										neighborX,
										neighborY,
										neighborZ,
										neighborWSB,
										level,
										timestep,
										numberOfTimesteps,
										numberOfParticles,
										size_Mat,
										isEvenTimestep);
      getLastCudaError("MoveParticles execution failed");
}
//////////////////////////////////////////////////////////////////////////
void initRandomDevice(hiprandState* state,
								 unsigned int size_Mat,
								 unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);
   initRandom<<< grid.grid, grid.threads >>> (state);
   getLastCudaError("initRandom execution failed");
}
//////////////////////////////////////////////////////////////////////////
void generateRandomValuesDevice( hiprandState* state,
											unsigned int size_Mat,
											real* randArray,
											unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);
   generateRandomValues<<< grid.grid, grid.threads >>> (state,randArray);
   getLastCudaError("generateRandomValues execution failed");
}
//////////////////////////////////////////////////////////////////////////
void CalcTurbulenceIntensityDevice(
   real* vxx,
   real* vyy,
   real* vzz,
   real* vxy,
   real* vxz,
   real* vyz,
   real* vx_mean,
   real* vy_mean,
   real* vz_mean,
   real* DD,
   uint* typeOfGridNode,
   unsigned int* neighborX,
   unsigned int* neighborY,
   unsigned int* neighborZ,
   unsigned int size_Mat,
   bool isEvenTimestep,
   uint numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);
   CalcTurbulenceIntensity<<<grid.grid, grid.threads>>>(
     vxx,
     vyy,
     vzz,
	 vxy,
     vxz,
     vyz,
     vx_mean,
     vy_mean,
     vz_mean,
     DD,
     typeOfGridNode,
     neighborX,
     neighborY,
     neighborZ,
     size_Mat,
     isEvenTimestep);

   getLastCudaError("CalcTurbulenceIntensity execution failed");
}













