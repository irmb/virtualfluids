//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ /
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
// includes, cuda
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "LBM/LB.h"
#include "CudaGrid.h"

// includes, kernels
#include "GPU/GPU_Kernels.cuh"
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelCas27( unsigned int grid_nx,
                             unsigned int grid_ny,
                             unsigned int grid_nz,
                             real s9,
                             unsigned int* bcMatD,
                             unsigned int* neighborX,
                             unsigned int* neighborY,
                             unsigned int* neighborZ,
                             real* DD,
                             int size_Mat,
                             bool EvenOrOdd)
{
   dim3 threads       ( grid_nx, 1, 1 );
   dim3 grid          ( grid_ny, grid_nz );   // Gitter fuer Kollision und Propagation

      LB_Kernel_Casc27<<< grid, threads >>>( s9,
                                             bcMatD,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             DD,
                                             size_Mat,
                                             EvenOrOdd);
     getLastCudaError("LB_Kernel_Casc27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelCasSP27( unsigned int numberOfThreads,
                               real s9,
                               unsigned int* bcMatD,
                               unsigned int* neighborX,
                               unsigned int* neighborY,
                               unsigned int* neighborZ,
                               real* DD,
                               int size_Mat,
                               bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_Casc_SP_27<<< grid.grid, grid.threads >>>(s9,
                                                bcMatD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                DD,
                                                size_Mat,
                                                EvenOrOdd);
      getLastCudaError("LB_Kernel_Casc_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelCasSPMS27( unsigned int numberOfThreads,
                                 real s9,
                                 unsigned int* bcMatD,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 real* DD,
                                 int size_Mat,
                                 bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_Casc_SP_MS_27<<< grid.grid, grid.threads >>>(s9,
                                                   bcMatD,
                                                   neighborX,
                                                   neighborY,
                                                   neighborZ,
                                                   DD,
                                                   size_Mat,
                                                   EvenOrOdd);
      getLastCudaError("LB_Kernel_Casc_SP_MS_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelCasSPMSOHM27( unsigned int numberOfThreads,
                                    real s9,
                                    unsigned int* bcMatD,
                                    unsigned int* neighborX,
                                    unsigned int* neighborY,
                                    unsigned int* neighborZ,
                                    real* DD,
                                    int size_Mat,
                                    bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_Casc_SP_MS_OHM_27<<< grid.grid, grid.threads >>>(  s9,
                                                         bcMatD,
                                                         neighborX,
                                                         neighborY,
                                                         neighborZ,
                                                         DD,
                                                         size_Mat,
                                                         EvenOrOdd);
      getLastCudaError("LB_Kernel_Casc_SP_MS_OHM_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelKumCompSRTSP27(
	unsigned int numberOfThreads,
	real omega,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* DDStart,
	int size_Mat,
	int level,
	real* forces,
	bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

   LB_Kernel_Kum_New_Comp_SRT_SP_27 <<< grid.grid, grid.threads >>>(
	   omega,
	   bcMatD,
	   neighborX,
	   neighborY,
	   neighborZ,
	   DDStart,
	   size_Mat,
	   level,
	   forces,
	   EvenOrOdd);
      getLastCudaError("LB_Kernel_Kum_New_Comp_SRT_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelKum1hSP27(    unsigned int numberOfThreads,
									real omega,
									real deltaPhi,
									real angularVelocity,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* coordX,
									real* coordY,
									real* coordZ,
									real* DDStart,
									int size_Mat,
									bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

		LB_Kernel_Kum_1h_SP_27<<< grid.grid, grid.threads >>>(omega,
													deltaPhi,
													angularVelocity,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													coordX,
													coordY,
													coordZ,
													DDStart,
													size_Mat,
													EvenOrOdd);
		getLastCudaError("LB_Kernel_Kum_New_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelCascadeSP27(  unsigned int numberOfThreads,
									real s9,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* DD,
									int size_Mat,
									bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

		LB_Kernel_Cascade_SP_27<<< grid.grid, grid.threads >>>(s9,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													size_Mat,
													EvenOrOdd);
		getLastCudaError("LB_Kernel_Cascade_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelKumNewSP27(   unsigned int numberOfThreads,
									real s9,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* DD,
									int size_Mat,
									bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


		LB_Kernel_Kum_New_SP_27<<< grid.grid, grid.threads >>>(s9,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													size_Mat,
													EvenOrOdd);
		getLastCudaError("LB_Kernel_Kum_New_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelKumNewCompSP27(unsigned int numberOfThreads,
									real s9,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* DD,
									int size_Mat,
									int size_Array,
									int level,
									real* forces,
									bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

		//LB_Kernel_Kum_New_Comp_SP_27<<< grid.grid, grid.threads >>>(	s9,
		//													bcMatD,
		//													neighborX,
		//													neighborY,
		//													neighborZ,
		//													DD,
		//													size_Mat,
		//													level,
		//													forces,
		//													EvenOrOdd);
		//getLastCudaError("LB_Kernel_Kum_New_Comp_SP_27 execution failed");
}

//////////////////////////////////////////////////////////////////////////
extern "C" void CumulantOnePreconditionedErrorDiffusionChimCompSP27(unsigned int numberOfThreads,
																	real s9,
																	unsigned int* bcMatD,
																	unsigned int* neighborX,
																	unsigned int* neighborY,
																	unsigned int* neighborZ,
																	real* DD,
																	int size_Mat,
																	int size_Array,
																	int level,
																	real* forces,
																	bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	Cumulant_One_preconditioned_errorDiffusion_chim_Comp_SP_27 <<< grid.grid, grid.threads >>>(	s9,
																						bcMatD,
																						neighborX,
																						neighborY,
																						neighborZ,
																						DD,
																						size_Mat,
																						level,
																						forces,
																						EvenOrOdd);
		getLastCudaError("Cumulant_One_preconditioned_chim_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CumulantOnePreconditionedChimCompSP27(  unsigned int numberOfThreads,
														real s9,
														unsigned int* bcMatD,
														unsigned int* neighborX,
														unsigned int* neighborY,
														unsigned int* neighborZ,
														real* DD,
														int size_Mat,
														int size_Array,
														int level,
														real* forces,
														bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	Cumulant_One_preconditioned_chim_Comp_SP_27 <<< grid.grid, grid.threads >>>(	s9,
																		bcMatD,
																		neighborX,
																		neighborY,
																		neighborZ,
																		DD,
																		size_Mat,
																		level,
																		forces,
																		EvenOrOdd);
		getLastCudaError("Cumulant_One_preconditioned_chim_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CumulantOneChimCompSP27(unsigned int numberOfThreads,
										real s9,
										unsigned int* bcMatD,
										unsigned int* neighborX,
										unsigned int* neighborY,
										unsigned int* neighborZ,
										real* DD,
										int size_Mat,
										int size_Array,
										int level,
										real* forces,
										bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	Cumulant_One_chim_Comp_SP_27 <<< grid.grid, grid.threads >>>(	s9,
														bcMatD,
														neighborX,
														neighborY,
														neighborZ,
														DD,
														size_Mat,
														level,
														forces,
														EvenOrOdd);
		getLastCudaError("Cumulant_One_chim_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelKumIsoTestSP27(unsigned int numberOfThreads,
									 real s9,
									 unsigned int* bcMatD,
									 unsigned int* neighborX,
									 unsigned int* neighborY,
									 unsigned int* neighborZ,
									 real* DD,
									 real* dxxUx,
									 real* dyyUy,
									 real* dzzUz,
									 int size_Mat,
									 bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	LB_Kernel_Kum_IsoTest_SP_27<<< grid.grid, grid.threads >>>(s9,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													dxxUx,
													dyyUy,
													dzzUz,
													size_Mat,
													EvenOrOdd);
	getLastCudaError("LB_Kernel_Kum_IsoTest_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelKumCompSP27(  unsigned int numberOfThreads,
									real s9,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									real* DD,
									int size_Mat,
									bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


		LB_Kernel_Kum_Comp_SP_27<<< grid.grid, grid.threads >>>(s9,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													size_Mat,
													EvenOrOdd);
		getLastCudaError("LB_Kernel_Kum_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelPMCumOneCompSP27(unsigned int numberOfThreads,
									   real omega,
									   unsigned int* neighborX,
									   unsigned int* neighborY,
									   unsigned int* neighborZ,
									   real* DD,
									   int size_Mat,
									   int level,
									   real* forces,
									   real porosity,
									   real darcy,
									   real forchheimer,
									   unsigned int sizeOfPorousMedia,
									   unsigned int* nodeIdsPorousMedia,
									   bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	LB_Kernel_PM_Cum_One_Comp_SP_27 <<< grid.grid, grid.threads >>>(omega,
														  neighborX,
														  neighborY,
														  neighborZ,
														  DD,
														  size_Mat,
														  level,
														  forces,
														  porosity,
														  darcy,
														  forchheimer,
														  sizeOfPorousMedia,
														  nodeIdsPorousMedia,
														  EvenOrOdd);
	getLastCudaError("LB_Kernel_PM_Cum_One_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelWaleBySoniMalavCumAA2016CompSP27(
	unsigned int numberOfThreads,
	real s9,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int* neighborWSB,
	real* veloX,
	real* veloY,
	real* veloZ,
	real* DD,
	real* turbulentViscosity,
	int size_Mat,
	int size_Array,
	int level,
	real* forces,
	bool EvenOrOdd)
{
	//int Grid = size_Array / numberOfThreads;
	//dim3 grid(Grid, 1, 1);
	//dim3 threads(numberOfThreads, 1, 1 );

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	LB_Kernel_WaleBySoniMalav_Cum_AA2016_Comp_SP_27 << < grid.grid, grid.threads >> >(
		s9,
		bcMatD,
		neighborX,
		neighborY,
		neighborZ,
		neighborWSB,
		veloX,
		veloY,
		veloZ,
		DD,
		turbulentViscosity,
		size_Mat,
		level,
		forces,
		EvenOrOdd);
	getLastCudaError("LB_Kernel_WaleBySoniMalav_Cum_AA2016_Comp_SP_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelADincomp7(   unsigned int numberOfThreads,
								   real diffusivity,
								   unsigned int* bcMatD,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   real* DD,
								   real* DD7,
								   int size_Mat,
								   bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_AD_Incomp_7<<< grid.grid, grid.threads >>>( diffusivity,
												  bcMatD,
												  neighborX,
												  neighborY,
												  neighborZ,
												  DD,
												  DD7,
												  size_Mat,
												  EvenOrOdd);
      getLastCudaError("LB_Kernel_AD_Incomp_7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void KernelADincomp27( unsigned int numberOfThreads,
								  real diffusivity,
								  unsigned int* bcMatD,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  real* DD,
								  real* DD27,
								  int size_Mat,
								  bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LB_Kernel_AD_Incomp_27<<< grid.grid, grid.threads >>>( diffusivity,
													bcMatD,
													neighborX,
													neighborY,
													neighborZ,
													DD,
													DD27,
													size_Mat,
													EvenOrOdd);
      getLastCudaError("LB_Kernel_AD_Incomp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void Init27( int myid,
                        int numprocs,
                        real u0,
                        unsigned int* geoD,
                        unsigned int* neighborX,
                        unsigned int* neighborY,
                        unsigned int* neighborZ,
                        real* vParab,
                        unsigned int size_Mat,
                        unsigned int grid_nx,
                        unsigned int grid_ny,
                        unsigned int grid_nz,
                        real* DD,
                        int level,
                        int maxlevel)
{
   dim3 threads       ( grid_nx, 1, 1 );
   dim3 grid          ( grid_ny, grid_nz );   // Gitter fuer Kollision und Propagation

      LBInit27<<< grid, threads >>> (  myid,
                                       numprocs,
                                       u0,
                                       geoD,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       vParab,
                                       size_Mat,
                                       grid_nx,
                                       grid_ny,
                                       grid_nz,
                                       DD,
                                       level,
                                       maxlevel);
      getLastCudaError("LBInit27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void InitNonEqPartSP27( unsigned int numberOfThreads,
                                   unsigned int* neighborX,
                                   unsigned int* neighborY,
                                   unsigned int* neighborZ,
                                   unsigned int* neighborWSB,
                                   unsigned int* geoD,
                                   real* rho,
                                   real* ux,
                                   real* uy,
                                   real* uz,
                                   unsigned int size_Mat,
                                   real* DD,
                                   real omega,
                                   bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBInitNonEqPartSP27<<< grid.grid, grid.threads >>>( neighborX,
                                                neighborY,
                                                neighborZ,
                                                neighborWSB,
                                                geoD,
                                                rho,
                                                ux,
                                                uy,
                                                uz,
                                                size_Mat,
                                                DD,
                                                omega,
                                                EvenOrOdd);
      getLastCudaError("LBInitNonEqPartSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void InitThS7(     unsigned int numberOfThreads,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int* geoD,
                              real* Conc,
                              real* ux,
                              real* uy,
                              real* uz,
                              unsigned int size_Mat,
                              real* DD7,
                              bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      InitAD7<<< grid.grid, grid.threads >>>( neighborX,
                                       neighborY,
                                       neighborZ,
                                       geoD,
                                       Conc,
                                       ux,
                                       uy,
                                       uz,
                                       size_Mat,
                                       DD7,
                                       EvenOrOdd);
      getLastCudaError("InitAD7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void InitADDev27( unsigned int numberOfThreads,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int* geoD,
                           real* Conc,
                           real* ux,
                           real* uy,
                           real* uz,
                           unsigned int size_Mat,
                           real* DD27,
                           bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      InitAD27<<< grid.grid, grid.threads >>>(neighborX,
                                       neighborY,
                                       neighborZ,
                                       geoD,
                                       Conc,
                                       ux,
                                       uy,
                                       uz,
                                       size_Mat,
                                       DD27,
                                       EvenOrOdd);
      getLastCudaError("InitAD27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void PostProcessorF3_2018Fehlberg(
	unsigned int numberOfThreads,
	real omega,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* rhoOut,
	real* vxOut,
	real* vyOut,
	real* vzOut,
	real* DDStart,
	real* G6,
	int size_Mat,
	int level,
	real* forces,
	bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	  LB_PostProcessor_F3_2018_Fehlberg <<< grid.grid, grid.threads >>> (   omega,
																  bcMatD,
																  neighborX,
																  neighborY,
																  neighborZ,
																  rhoOut,
																  vxOut,
																  vyOut,
																  vzOut,
																  DDStart,
																  G6,
																  size_Mat,
																  level,
																  forces,
																  EvenOrOdd);
      getLastCudaError("LB_PostProcessor_F3_2018_Fehlberg execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcMac27( real* vxD,
                           real* vyD,
                           real* vzD,
                           real* rhoD,
                           unsigned int* geoD,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int size_Mat,
                           unsigned int grid_nx,
                           unsigned int grid_ny,
                           unsigned int grid_nz,
                           real* DD,
                           bool isEvenTimestep)
{
   dim3 threads       ( grid_nx, 1, 1 );
   dim3 grid          ( grid_ny, grid_nz );

      LBCalcMac27<<< grid, threads >>> (  vxD,
                                          vyD,
                                          vzD,
                                          rhoD,
                                          geoD,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          DD,
                                          isEvenTimestep);
      getLastCudaError("LBCalcMac27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcMacSP27( real* vxD,
                             real* vyD,
                             real* vzD,
                             real* rhoD,
                             real* pressD,
                             unsigned int* geoD,
                             unsigned int* neighborX,
                             unsigned int* neighborY,
                             unsigned int* neighborZ,
                             unsigned int size_Mat,
                             unsigned int numberOfThreads,
                             real* DD,
                             bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalcMacSP27<<< grid.grid, grid.threads >>> (   vxD,
                                             vyD,
                                             vzD,
                                             rhoD,
                                             pressD,
                                             geoD,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             DD,
                                             isEvenTimestep);
      getLastCudaError("LBCalcMacSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcMacCompSP27( real* vxD,
								 real* vyD,
								 real* vzD,
								 real* rhoD,
								 real* pressD,
								 unsigned int* geoD,
								 unsigned int* neighborX,
								 unsigned int* neighborY,
								 unsigned int* neighborZ,
								 unsigned int size_Mat,
								 unsigned int numberOfThreads,
								 real* DD,
								 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalcMacCompSP27<<< grid.grid, grid.threads >>> (   vxD,
												 vyD,
												 vzD,
												 rhoD,
												 pressD,
												 geoD,
												 neighborX,
												 neighborY,
												 neighborZ,
												 size_Mat,
												 DD,
												 isEvenTimestep);
      getLastCudaError("LBCalcMacSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcMacThS7(  real* Conc,
                              unsigned int* geoD,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              unsigned int numberOfThreads,
                              real* DD7,
                              bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      CalcConc7<<< grid.grid, grid.threads >>> (Conc,
                                          geoD,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          DD7,
                                          isEvenTimestep);
      getLastCudaError("CalcConc7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void PlaneConcThS7(real* Conc,
							  int* kPC,
							  unsigned int numberOfPointskPC,
							  unsigned int* geoD,
							  unsigned int* neighborX,
							  unsigned int* neighborY,
							  unsigned int* neighborZ,
							  unsigned int size_Mat,
                              unsigned int numberOfThreads,
							  real* DD7,
							  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfPointskPC);

      GetPlaneConc7<<< grid.grid, grid.threads >>> (	Conc,
												kPC,
												numberOfPointskPC,
												geoD,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												DD7,
												isEvenTimestep);
      getLastCudaError("GetPlaneConc7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void PlaneConcThS27(real* Conc,
							   int* kPC,
							   unsigned int numberOfPointskPC,
							   unsigned int* geoD,
							   unsigned int* neighborX,
							   unsigned int* neighborY,
							   unsigned int* neighborZ,
							   unsigned int size_Mat,
                               unsigned int numberOfThreads,
							   real* DD27,
							   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfPointskPC);

      GetPlaneConc27<<< grid.grid, grid.threads >>> (	Conc,
												kPC,
												numberOfPointskPC,
												geoD,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												DD27,
												isEvenTimestep);
      getLastCudaError("GetPlaneConc27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcConcentration27( unsigned int numberOfThreads,
                                     real* Conc,
                                     unsigned int* geoD,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     real* DD27,
                                     bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      CalcConc27<<< grid.grid, grid.threads >>> (  Conc,
                                             geoD,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             DD27,
                                             isEvenTimestep);
      getLastCudaError("CalcConc27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcMedSP27(  real* vxD,
                              real* vyD,
                              real* vzD,
                              real* rhoD,
                              real* pressD,
                              unsigned int* geoD,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              unsigned int numberOfThreads,
                              real* DD,
                              bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalcMedSP27<<< grid.grid, grid.threads >>> (   vxD,
                                             vyD,
                                             vzD,
                                             rhoD,
                                             pressD,
                                             geoD,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             DD,
                                             isEvenTimestep);
      getLastCudaError("LBCalcMedSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcMedCompSP27(  real* vxD,
								  real* vyD,
								  real* vzD,
								  real* rhoD,
								  real* pressD,
								  unsigned int* geoD,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  unsigned int size_Mat,
								  unsigned int numberOfThreads,
								  real* DD,
								  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalcMedCompSP27<<< grid.grid, grid.threads >>> (   vxD,
												 vyD,
												 vzD,
												 rhoD,
												 pressD,
												 geoD,
												 neighborX,
												 neighborY,
												 neighborZ,
												 size_Mat,
												 DD,
												 isEvenTimestep);
      getLastCudaError("LBCalcMedSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcMedCompAD27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	real* concD,
	unsigned int* geoD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	unsigned int numberOfThreads,
	real* DD,
	real* DD_AD,
	bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBCalcMedCompAD27 <<< grid.grid, grid.threads >>> (
		vxD,
		vyD,
		vzD,
		rhoD,
		pressD,
		concD,
		geoD,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		DD,
		DD_AD,
		isEvenTimestep);
	getLastCudaError("LBCalcMedAD27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcMacMedSP27(  real* vxD,
                                 real* vyD,
                                 real* vzD,
                                 real* rhoD,
                                 real* pressD,
                                 unsigned int* geoD,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 unsigned int tdiff,
                                 unsigned int size_Mat,
                                 unsigned int numberOfThreads,
                                 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalcMacMedSP27<<< grid.grid, grid.threads >>> (   vxD,
                                                vyD,
                                                vzD,
                                                rhoD,
                                                pressD,
                                                geoD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                tdiff,
                                                size_Mat,
                                                isEvenTimestep);
      getLastCudaError("LBCalcMacMedSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ResetMedianValuesSP27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	unsigned int size_Mat,
	unsigned int numberOfThreads,
	bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);


	LBResetMedianValuesSP27 << < grid.grid, grid.threads >> > (
		vxD,
		vyD,
		vzD,
		rhoD,
		pressD,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("LBResetMedianValuesSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ResetMedianValuesAD27(
	real* vxD,
	real* vyD,
	real* vzD,
	real* rhoD,
	real* pressD,
	real* concD,
	unsigned int size_Mat,
	unsigned int numberOfThreads,
	bool isEvenTimestep)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

	LBResetMedianValuesAD27 << < grid.grid, grid.threads >> > (
		vxD,
		vyD,
		vzD,
		rhoD,
		pressD,
		concD,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("LBResetMedianValuesAD27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void Calc2ndMomentsIncompSP27(real* kxyFromfcNEQ,
										 real* kyzFromfcNEQ,
										 real* kxzFromfcNEQ,
										 real* kxxMyyFromfcNEQ,
										 real* kxxMzzFromfcNEQ,
										 unsigned int* geoD,
										 unsigned int* neighborX,
										 unsigned int* neighborY,
										 unsigned int* neighborZ,
										 unsigned int size_Mat,
										 unsigned int numberOfThreads,
										 real* DD,
										 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalc2ndMomentsIncompSP27<<< grid.grid, grid.threads >>> (  kxyFromfcNEQ,
														 kyzFromfcNEQ,
														 kxzFromfcNEQ,
														 kxxMyyFromfcNEQ,
														 kxxMzzFromfcNEQ,
														 geoD,
														 neighborX,
														 neighborY,
														 neighborZ,
														 size_Mat,
														 DD,
														 isEvenTimestep);
      getLastCudaError("LBCalc2ndMomentsIncompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void Calc2ndMomentsCompSP27( real* kxyFromfcNEQ,
										real* kyzFromfcNEQ,
										real* kxzFromfcNEQ,
										real* kxxMyyFromfcNEQ,
										real* kxxMzzFromfcNEQ,
										unsigned int* geoD,
										unsigned int* neighborX,
										unsigned int* neighborY,
										unsigned int* neighborZ,
										unsigned int size_Mat,
										unsigned int numberOfThreads,
										real* DD,
										bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalc2ndMomentsCompSP27<<< grid.grid, grid.threads >>> (kxyFromfcNEQ,
													 kyzFromfcNEQ,
													 kxzFromfcNEQ,
													 kxxMyyFromfcNEQ,
													 kxxMzzFromfcNEQ,
													 geoD,
													 neighborX,
													 neighborY,
													 neighborZ,
													 size_Mat,
													 DD,
													 isEvenTimestep);
      getLastCudaError("LBCalc2ndMomentsCompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void Calc3rdMomentsIncompSP27(real* CUMbbb,
										 real* CUMabc,
										 real* CUMbac,
										 real* CUMbca,
										 real* CUMcba,
										 real* CUMacb,
										 real* CUMcab,
										 unsigned int* geoD,
										 unsigned int* neighborX,
										 unsigned int* neighborY,
										 unsigned int* neighborZ,
										 unsigned int size_Mat,
										 unsigned int numberOfThreads,
										 real* DD,
										 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalc3rdMomentsIncompSP27<<< grid.grid, grid.threads >>> (  CUMbbb,
														 CUMabc,
														 CUMbac,
														 CUMbca,
														 CUMcba,
														 CUMacb,
														 CUMcab,
														 geoD,
														 neighborX,
														 neighborY,
														 neighborZ,
														 DD,
														 size_Mat,
														 isEvenTimestep);
      getLastCudaError("LBCalc3rdMomentsIncompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void Calc3rdMomentsCompSP27( real* CUMbbb,
										real* CUMabc,
										real* CUMbac,
										real* CUMbca,
										real* CUMcba,
										real* CUMacb,
										real* CUMcab,
										unsigned int* geoD,
										unsigned int* neighborX,
										unsigned int* neighborY,
										unsigned int* neighborZ,
										unsigned int size_Mat,
										unsigned int numberOfThreads,
										real* DD,
										bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalc3rdMomentsCompSP27<<< grid.grid, grid.threads >>> (CUMbbb,
													 CUMabc,
													 CUMbac,
													 CUMbca,
													 CUMcba,
													 CUMacb,
													 CUMcab,
													 geoD,
													 neighborX,
													 neighborY,
													 neighborZ,
													 DD,
													 size_Mat,
													 isEvenTimestep);
      getLastCudaError("LBCalc3rdMomentsCompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcHigherMomentsIncompSP27(real* CUMcbb,
											real* CUMbcb,
											real* CUMbbc,
											real* CUMcca,
											real* CUMcac,
											real* CUMacc,
											real* CUMbcc,
											real* CUMcbc,
											real* CUMccb,
											real* CUMccc,
											unsigned int* geoD,
											unsigned int* neighborX,
											unsigned int* neighborY,
											unsigned int* neighborZ,
											unsigned int size_Mat,
											unsigned int numberOfThreads,
											real* DD,
											bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalcHigherMomentsIncompSP27<<< grid.grid, grid.threads >>> (CUMcbb,
														  CUMbcb,
														  CUMbbc,
														  CUMcca,
														  CUMcac,
														  CUMacc,
														  CUMbcc,
														  CUMcbc,
														  CUMccb,
														  CUMccc,
														  geoD,
														  neighborX,
														  neighborY,
														  neighborZ,
														  DD,
														  size_Mat,
														  isEvenTimestep);
      getLastCudaError("LBCalcHigherMomentsIncompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcHigherMomentsCompSP27(  real* CUMcbb,
											real* CUMbcb,
											real* CUMbbc,
											real* CUMcca,
											real* CUMcac,
											real* CUMacc,
											real* CUMbcc,
											real* CUMcbc,
											real* CUMccb,
											real* CUMccc,
											unsigned int* geoD,
											unsigned int* neighborX,
											unsigned int* neighborY,
											unsigned int* neighborZ,
											unsigned int size_Mat,
											unsigned int numberOfThreads,
											real* DD,
											bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);

      LBCalcHigherMomentsCompSP27<<< grid.grid, grid.threads >>> (  CUMcbb,
														  CUMbcb,
														  CUMbbc,
														  CUMcca,
														  CUMcac,
														  CUMacc,
														  CUMbcc,
														  CUMcbc,
														  CUMccb,
														  CUMccc,
														  geoD,
														  neighborX,
														  neighborY,
														  neighborZ,
														  DD,
														  size_Mat,
														  isEvenTimestep);
      getLastCudaError("LBCalcHigherMomentsCompSP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void LBCalcMeasurePoints27(real* vxMP,
                                      real* vyMP,
                                      real* vzMP,
                                      real* rhoMP,
                                      unsigned int* kMP,
                                      unsigned int numberOfPointskMP,
                                      unsigned int MPClockCycle,
                                      unsigned int t,
                                      unsigned int* geoD,
                                      unsigned int* neighborX,
                                      unsigned int* neighborY,
                                      unsigned int* neighborZ,
                                      unsigned int size_Mat,
                                      real* DD,
                                      unsigned int numberOfThreads,
                                      bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfPointskMP);

      LBCalcMeasurePoints<<< grid.grid, grid.threads >>> (vxMP,
                                                vyMP,
                                                vzMP,
                                                rhoMP,
                                                kMP,
                                                numberOfPointskMP,
                                                MPClockCycle,
                                                t,
                                                geoD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                DD,
                                                isEvenTimestep);
      getLastCudaError("LBCalcMeasurePoints execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void BcPress27( int nx,
                           int ny,
                           int tz,
                           unsigned int grid_nx,
                           unsigned int grid_ny,
                           unsigned int* bcMatD,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           real* DD,
                           unsigned int size_Mat,
                           bool isEvenTimestep)
{
   dim3 threads       ( grid_nx, 1, 1 );
   dim3 grid          ( grid_ny, 1 );

      LB_BC_Press_East27<<< grid, threads >>> ( nx,
                                                ny,
                                                tz,
                                                bcMatD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                DD,
                                                size_Mat,
                                                isEvenTimestep);
      getLastCudaError("LB_BC_Press_East27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void BcVel27(int nx,
                        int ny,
                        int nz,
                        int itz,
                        unsigned int grid_nx,
                        unsigned int grid_ny,
                        unsigned int* bcMatD,
                        unsigned int* neighborX,
                        unsigned int* neighborY,
                        unsigned int* neighborZ,
                        real* DD,
                        unsigned int size_Mat,
                        bool isEvenTimestep,
                        real u0x,
                        real om)
{
   dim3 threads       ( grid_nx, 1, 1 );
   dim3 grid          ( grid_ny, 1 );

      LB_BC_Vel_West_27<<< grid, threads >>> (  nx,
                                                ny,
                                                nz,
                                                itz,
                                                bcMatD,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                DD,
                                                size_Mat,
                                                isEvenTimestep,
                                                u0x,
                                                grid_nx,
                                                grid_ny,
                                                om);
      getLastCudaError("LB_BC_Vel_West_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADPressDev7( unsigned int numberOfThreads,
                              int nx,
                              int ny,
                              real* DD,
                              real* DD7,
                              real* temp,
                              real* velo,
                              real diffusivity,
                              int* k_Q,
                              real* QQ,
                              unsigned int sizeQ,
                              unsigned int numberOfBCnodes,
                              real om1,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      QADPress7<<< grid.grid, grid.threads >>>( nx,
                                       ny,
                                       DD,
                                       DD7,
                                       temp,
                                       velo,
                                       diffusivity,
                                       k_Q,
                                       QQ,
                                       sizeQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
      getLastCudaError("QADPress7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADPressDev27(unsigned int numberOfThreads,
                              int nx,
                              int ny,
                              real* DD,
                              real* DD27,
                              real* temp,
                              real* velo,
                              real diffusivity,
                              int* k_Q,
                              real* QQ,
                              unsigned int sizeQ,
                              unsigned int numberOfBCnodes,
                              real om1,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      QADPress27<<< grid.grid, grid.threads >>>(   nx,
                                          ny,
                                          DD,
                                          DD27,
                                          temp,
                                          velo,
                                          diffusivity,
                                          k_Q,
                                          QQ,
                                          sizeQ,
                                          numberOfBCnodes,
                                          om1,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          isEvenTimestep);
      getLastCudaError("QADPress27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADPressNEQNeighborDev27(
											unsigned int numberOfThreads,
											real* DD,
											real* DD27,
											int* k_Q,
											int* k_N,
											int numberOfBCnodes,
											unsigned int* neighborX,
											unsigned int* neighborY,
											unsigned int* neighborZ,
											unsigned int size_Mat,
											bool isEvenTimestep
										)
{

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


   QADPressNEQNeighbor27<<< grid.grid, grid.threads >>>(
												DD,
												DD27,
												k_Q,
												k_N,
												numberOfBCnodes,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep
											  );
   getLastCudaError("QADPressNEQNeighbor27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADVelDev7(unsigned int numberOfThreads,
                           int nx,
                           int ny,
                           real* DD,
                           real* DD7,
                           real* temp,
                           real* velo,
                           real diffusivity,
                           int* k_Q,
                           real* QQ,
                           unsigned int sizeQ,
                           unsigned int numberOfBCnodes,
                           real om1,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int size_Mat,
                           bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      QADVel7<<< grid.grid, grid.threads >>> (  nx,
                                       ny,
                                       DD,
                                       DD7,
                                       temp,
                                       velo,
                                       diffusivity,
                                       k_Q,
                                       QQ,
                                       sizeQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
      getLastCudaError("QADVel7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADVelDev27(  unsigned int numberOfThreads,
                              int nx,
                              int ny,
                              real* DD,
                              real* DD27,
                              real* temp,
                              real* velo,
                              real diffusivity,
                              int* k_Q,
                              real* QQ,
                              unsigned int sizeQ,
                              unsigned int numberOfBCnodes,
                              real om1,
                              unsigned int* neighborX,
                              unsigned int* neighborY,
                              unsigned int* neighborZ,
                              unsigned int size_Mat,
                              bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      QADVel27<<< grid.grid, grid.threads >>> (nx,
                                      ny,
                                      DD,
                                      DD27,
                                      temp,
                                      velo,
                                      diffusivity,
                                      k_Q,
                                      QQ,
                                      sizeQ,
                                      numberOfBCnodes,
                                      om1,
                                      neighborX,
                                      neighborY,
                                      neighborZ,
                                      size_Mat,
                                      isEvenTimestep);
      getLastCudaError("QADVel27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADDev7(unsigned int numberOfThreads,
                        int nx,
                        int ny,
                        real* DD,
                        real* DD7,
                        real* temp,
                        real diffusivity,
                        int* k_Q,
                        real* QQ,
                        unsigned int sizeQ,
                        unsigned int numberOfBCnodes,
                        real om1,
                        unsigned int* neighborX,
                        unsigned int* neighborY,
                        unsigned int* neighborZ,
                        unsigned int size_Mat,
                        bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      QAD7<<< grid.grid, grid.threads >>> (     nx,
                                       ny,
                                       DD,
                                       DD7,
                                       temp,
                                       diffusivity,
                                       k_Q,
                                       QQ,
                                       sizeQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
      getLastCudaError("QAD7 execution failed");
}


//////////////////////////////////////////////////////////////////////////
// Other advection diffusion kernels are in kernel factory :(
extern "C" void FactorizedCentralMomentsAdvectionDiffusionDeviceKernel(
   uint numberOfThreads,
   real omegaDiffusivity,
   uint* typeOfGridNode,
   uint* neighborX,
   uint* neighborY,
   uint* neighborZ,
   real* distributions,
   real* distributionsAD,
   int size_Mat,
   real* forces,
   bool isEvenTimestep)
{
   int Grid = (size_Mat / numberOfThreads) + 1;
   dim3 grid(Grid, 1, 1);
   dim3 threads(numberOfThreads, 1, 1);

   Factorized_Central_Moments_Advection_Diffusion_Device_Kernel <<< grid, threads >>> (
      omegaDiffusivity,
      typeOfGridNode,
      neighborX,
      neighborY,
      neighborZ,
      distributions,
      distributionsAD,
      size_Mat,
      forces,
      isEvenTimestep);
   getLastCudaError("Factorized_Central_Moments_Advection_Diffusion_Device_Kernel execution failed");
}

//////////////////////////////////////////////////////////////////////////
extern "C" void ADSlipVelDevComp(
	uint numberOfThreads,
	real * normalX,
	real * normalY,
	real * normalZ,
	real * distributions,
	real * distributionsAD,
	int* QindexArray,
	real * Qarrays,
	uint numberOfQs,
	real omegaDiffusivity,
	uint * neighborX,
	uint * neighborY,
	uint * neighborZ,
	uint size_Mat,
	bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfQs);

	AD_SlipVelDeviceComp << < grid.grid, grid.threads >> > (
		normalX,
		normalY,
		normalZ,
		distributions,
		distributionsAD,
		QindexArray,
		Qarrays,
		numberOfQs,
		omegaDiffusivity,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("AD_SlipVelDeviceComp execution failed");
}
//////////////////////////////////////////////////////////////////////////

extern "C" void QADDirichletDev27( unsigned int numberOfThreads,
								   int nx,
								   int ny,
								   real* DD,
								   real* DD27,
								   real* temp,
								   real diffusivity,
								   int* k_Q,
								   real* QQ,
								   unsigned int sizeQ,
								   unsigned int numberOfBCnodes,
								   real om1,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QADDirichlet27<<< grid.grid, grid.threads >>> (   nx,
											   ny,
											   DD,
											   DD27,
											   temp,
											   diffusivity,
											   k_Q,
											   QQ,
											   sizeQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QADDirichletDev27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADBBDev27(unsigned int numberOfThreads,
                           int nx,
                           int ny,
                           real* DD,
                           real* DD27,
                           real* temp,
                           real diffusivity,
                           int* k_Q,
                           real* QQ,
                           unsigned int sizeQ,
                           unsigned int numberOfBCnodes,
                           real om1,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int size_Mat,
                           bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QADBB27<<< grid.grid, grid.threads >>> (  nx,
                                       ny,
                                       DD,
                                       DD27,
                                       temp,
                                       diffusivity,
                                       k_Q,
                                       QQ,
                                       sizeQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
      getLastCudaError("QADBB27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QNoSlipADincompDev7(unsigned int numberOfThreads,
									int nx,
									int ny,
									real* DD,
									real* DD7,
									real* temp,
									real diffusivity,
									int* k_Q,
									real* QQ,
									unsigned int sizeQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QNoSlipADincomp7<<< grid.grid, grid.threads >>> ( nx,
											   ny,
											   DD,
											   DD7,
											   temp,
											   diffusivity,
											   k_Q,
											   QQ,
											   sizeQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QNoSlipADincomp7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QNoSlipADincompDev27(  unsigned int numberOfThreads,
									   int nx,
									   int ny,
									   real* DD,
									   real* DD27,
									   real* temp,
									   real diffusivity,
									   int* k_Q,
									   real* QQ,
									   unsigned int sizeQ,
									   unsigned int numberOfBCnodes,
									   real om1,
									   unsigned int* neighborX,
									   unsigned int* neighborY,
									   unsigned int* neighborZ,
									   unsigned int size_Mat,
									   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QNoSlipADincomp27<<< grid.grid, grid.threads >>> (nx,
											   ny,
											   DD,
											   DD27,
											   temp,
											   diffusivity,
											   k_Q,
											   QQ,
											   sizeQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QNoSlipADincomp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADVeloIncompDev7( unsigned int numberOfThreads,
								   int nx,
								   int ny,
								   real* DD,
								   real* DD7,
								   real* temp,
								   real* velo,
								   real diffusivity,
								   int* k_Q,
								   real* QQ,
								   unsigned int sizeQ,
								   unsigned int numberOfBCnodes,
								   real om1,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QADVeloIncomp7<<< grid.grid, grid.threads >>> (   nx,
											   ny,
											   DD,
											   DD7,
											   temp,
											   velo,
											   diffusivity,
											   k_Q,
											   QQ,
											   sizeQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QADVeloIncomp7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADVeloIncompDev27(   unsigned int numberOfThreads,
									  int nx,
									  int ny,
									  real* DD,
									  real* DD27,
									  real* temp,
									  real* velo,
									  real diffusivity,
									  int* k_Q,
									  real* QQ,
									  unsigned int sizeQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QADVeloIncomp27<<< grid.grid, grid.threads >>> ( nx,
											  ny,
											  DD,
											  DD27,
											  temp,
											  velo,
											  diffusivity,
											  k_Q,
											  QQ,
											  sizeQ,
											  numberOfBCnodes,
											  om1,
											  neighborX,
											  neighborY,
											  neighborZ,
											  size_Mat,
											  isEvenTimestep);
      getLastCudaError("QADVeloIncomp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADPressIncompDev7(   unsigned int numberOfThreads,
									  int nx,
									  int ny,
									  real* DD,
									  real* DD7,
									  real* temp,
									  real* velo,
									  real diffusivity,
									  int* k_Q,
									  real* QQ,
									  unsigned int sizeQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QADPressIncomp7<<< grid.grid, grid.threads >>>(   nx,
											   ny,
											   DD,
											   DD7,
											   temp,
											   velo,
											   diffusivity,
											   k_Q,
											   QQ,
											   sizeQ,
											   numberOfBCnodes,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QADPressIncomp7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QADPressIncompDev27(  unsigned int numberOfThreads,
									  int nx,
									  int ny,
									  real* DD,
									  real* DD27,
									  real* temp,
									  real* velo,
									  real diffusivity,
									  int* k_Q,
									  real* QQ,
									  unsigned int sizeQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QADPressIncomp27<<< grid.grid, grid.threads >>>( nx,
											  ny,
											  DD,
											  DD27,
											  temp,
											  velo,
											  diffusivity,
											  k_Q,
											  QQ,
											  sizeQ,
											  numberOfBCnodes,
											  om1,
											  neighborX,
											  neighborY,
											  neighborZ,
											  size_Mat,
											  isEvenTimestep);
      getLastCudaError("QADPressIncomp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QDev27( unsigned int numberOfThreads,
                        int nx,
                        int ny,
                        real* DD,
                        int* k_Q,
                        real* QQ,
                        unsigned int sizeQ,
                        unsigned int numberOfBCnodes,
                        real om1,
                        unsigned int* neighborX,
                        unsigned int* neighborY,
                        unsigned int* neighborZ,
                        unsigned int size_Mat,
                        bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QDevice27<<< grid.grid, grid.threads >>> (nx,
                                       ny,
                                       DD,
                                       k_Q,
                                       QQ,
                                       sizeQ,
                                       numberOfBCnodes,
                                       om1,
                                       neighborX,
                                       neighborY,
                                       neighborZ,
                                       size_Mat,
                                       isEvenTimestep);
      getLastCudaError("QDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QDevComp27( unsigned int numberOfThreads,
							int nx,
							int ny,
							real* DD,
							int* k_Q,
							real* QQ,
							unsigned int sizeQ,
							unsigned int numberOfBCnodes,
							real om1,
							unsigned int* neighborX,
							unsigned int* neighborY,
							unsigned int* neighborZ,
							unsigned int size_Mat,
							bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QDeviceComp27<<< grid.grid, grid.threads >>> (nx,
										   ny,
										   DD,
										   k_Q,
										   QQ,
										   sizeQ,
										   numberOfBCnodes,
										   om1,
										   neighborX,
										   neighborY,
										   neighborZ,
										   size_Mat,
										   isEvenTimestep);
      getLastCudaError("QDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QDevCompThinWalls27(unsigned int numberOfThreads,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int sizeQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* geom,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int* neighborWSB,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QDeviceCompThinWallsPartOne27 <<< grid.grid, grid.threads >>> (DD,
														 k_Q,
														 QQ,
														 sizeQ,
														 numberOfBCnodes,
														 om1,
														 neighborX,
														 neighborY,
														 neighborZ,
														 size_Mat,
														 isEvenTimestep);
   getLastCudaError("QDeviceCompThinWallsPartOne27 execution failed");

   QThinWallsPartTwo27 <<< grid.grid, grid.threads >>> ( DD,
												k_Q,
												QQ,
												sizeQ,
												numberOfBCnodes,
												geom,
												neighborX,
												neighborY,
												neighborZ,
												neighborWSB,
												size_Mat,
												isEvenTimestep);
   getLastCudaError("QThinWallsPartTwo27 execution failed");

}
//////////////////////////////////////////////////////////////////////////
extern "C" void QDev3rdMomentsComp27(   unsigned int numberOfThreads,
										int nx,
										int ny,
										real* DD,
										int* k_Q,
										real* QQ,
										unsigned int sizeQ,
										unsigned int numberOfBCnodes,
										real om1,
										unsigned int* neighborX,
										unsigned int* neighborY,
										unsigned int* neighborZ,
										unsigned int size_Mat,
										bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QDevice3rdMomentsComp27<<< grid.grid, grid.threads >>> (  nx,
													   ny,
													   DD,
													   k_Q,
													   QQ,
													   sizeQ,
													   numberOfBCnodes,
													   om1,
													   neighborX,
													   neighborY,
													   neighborZ,
													   size_Mat,
													   isEvenTimestep);
      getLastCudaError("QDevice3rdMomentsComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QDevIncompHighNu27( unsigned int numberOfThreads,
									int nx,
									int ny,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int sizeQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QDeviceIncompHighNu27<<< grid.grid, grid.threads >>> (nx,
												   ny,
												   DD,
												   k_Q,
												   QQ,
												   sizeQ,
												   numberOfBCnodes,
												   om1,
												   neighborX,
												   neighborY,
												   neighborZ,
												   size_Mat,
												   isEvenTimestep);
      getLastCudaError("QDeviceIncompHighNu27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QDevCompHighNu27(   unsigned int numberOfThreads,
									int nx,
									int ny,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int sizeQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QDeviceCompHighNu27<<< grid.grid, grid.threads >>> (  nx,
												   ny,
												   DD,
												   k_Q,
												   QQ,
												   sizeQ,
												   numberOfBCnodes,
												   om1,
												   neighborX,
												   neighborY,
												   neighborZ,
												   size_Mat,
												   isEvenTimestep);
      getLastCudaError("QDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDevicePlainBB27(unsigned int numberOfThreads,
									real* vx,
									real* vy,
									real* vz,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int sizeQ,
									int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDevPlainBB27<<< grid.grid, grid.threads >>> (  vx,
												vy,
												vz,
												DD,
												k_Q,
												QQ,
												sizeQ,
												numberOfBCnodes,
												om1,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep);
      getLastCudaError("QVelDevicePlainBB27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDeviceCouhette27(unsigned int numberOfThreads,
									real* vx,
									real* vy,
									real* vz,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int sizeQ,
									int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDevCouhette27<<< grid.grid, grid.threads >>> ( vx,
												vy,
												vz,
												DD,
												k_Q,
												QQ,
												sizeQ,
												numberOfBCnodes,
												om1,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep);
      getLastCudaError("QVelDevicePlainBB27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDevice1h27(   unsigned int numberOfThreads,
								  int nx,
								  int ny,
								  real* vx,
								  real* vy,
								  real* vz,
								  real* DD,
								  int* k_Q,
								  real* QQ,
								  unsigned int sizeQ,
								  unsigned int numberOfBCnodes,
								  real om1,
								  real Phi,
								  real angularVelocity,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  real* coordX,
								  real* coordY,
								  real* coordZ,
								  unsigned int size_Mat,
								  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDev1h27<<< grid.grid, grid.threads >>> (nx,
                                          ny,
                                          vx,
                                          vy,
                                          vz,
                                          DD,
                                          k_Q,
                                          QQ,
                                          sizeQ,
                                          numberOfBCnodes,
                                          om1,
										  Phi,
										  angularVelocity,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
										  coordX,
										  coordY,
										  coordZ,
                                          size_Mat,
                                          isEvenTimestep);
      getLastCudaError("QVelDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDev27(unsigned int numberOfThreads,
                          int nx,
                          int ny,
                          real* vx,
                          real* vy,
                          real* vz,
                          real* DD,
                          int* k_Q,
                          real* QQ,
                          unsigned int sizeQ,
                          unsigned int numberOfBCnodes,
                          real om1,
                          unsigned int* neighborX,
                          unsigned int* neighborY,
                          unsigned int* neighborZ,
                          unsigned int size_Mat,
                          bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDevice27<<< grid.grid, grid.threads >>> (nx,
                                          ny,
                                          vx,
                                          vy,
                                          vz,
                                          DD,
                                          k_Q,
                                          QQ,
                                          sizeQ,
                                          numberOfBCnodes,
                                          om1,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          isEvenTimestep);
      getLastCudaError("QVelDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDevCompPlusSlip27(unsigned int numberOfThreads,
									  int nx,
									  int ny,
									  real* vx,
									  real* vy,
									  real* vz,
									  real* DD,
									  int* k_Q,
									  real* QQ,
									  unsigned int sizeQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDeviceCompPlusSlip27<<< grid.grid, grid.threads >>> (nx,
													  ny,
													  vx,
													  vy,
													  vz,
													  DD,
													  k_Q,
													  QQ,
													  sizeQ,
													  numberOfBCnodes,
													  om1,
													  neighborX,
													  neighborY,
													  neighborZ,
													  size_Mat,
													  isEvenTimestep);
      getLastCudaError("QVelDeviceCompPlusSlip27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDevComp27(unsigned int numberOfThreads,
							  int nx,
							  int ny,
							  real* vx,
							  real* vy,
							  real* vz,
							  real* DD,
							  int* k_Q,
							  real* QQ,
							  unsigned int sizeQ,
							  unsigned int numberOfBCnodes,
							  real om1,
							  unsigned int* neighborX,
							  unsigned int* neighborY,
							  unsigned int* neighborZ,
							  unsigned int size_Mat,
							  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDeviceComp27<<< grid.grid, grid.threads >>> (nx,
											  ny,
											  vx,
											  vy,
											  vz,
											  DD,
											  k_Q,
											  QQ,
											  sizeQ,
											  numberOfBCnodes,
											  om1,
											  neighborX,
											  neighborY,
											  neighborZ,
											  size_Mat,
											  isEvenTimestep);
      getLastCudaError("QVelDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDevCompThinWalls27(unsigned int numberOfThreads,
							           real* vx,
							           real* vy,
							           real* vz,
							           real* DD,
							           int* k_Q,
							           real* QQ,
							           unsigned int sizeQ,
							           unsigned int numberOfBCnodes,
							           real om1,
									   unsigned int* geom,
							           unsigned int* neighborX,
							           unsigned int* neighborY,
							           unsigned int* neighborZ,
									   unsigned int* neighborWSB,
							           unsigned int size_Mat,
							           bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDeviceCompThinWallsPartOne27<<< grid.grid, grid.threads >>> (vx,
											                  vy,
											                  vz,
											                  DD,
											                  k_Q,
											                  QQ,
											                  sizeQ,
											                  numberOfBCnodes,
											                  om1,
											                  neighborX,
											                  neighborY,
											                  neighborZ,
											                  size_Mat,
											                  isEvenTimestep);
   getLastCudaError("QVelDeviceCompThinWallsPartOne27 execution failed");

	QThinWallsPartTwo27 <<< grid.grid, grid.threads >>> (DD,
											      k_Q,
											      QQ,
											      sizeQ,
											      numberOfBCnodes,
                                                  geom,
											      neighborX,
											      neighborY,
											      neighborZ,
                                                  neighborWSB,
											      size_Mat,
											      isEvenTimestep);
   getLastCudaError("QThinWallsPartTwo27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDevCompZeroPress27(   unsigned int numberOfThreads,
										  int nx,
										  int ny,
										  real* vx,
										  real* vy,
										  real* vz,
										  real* DD,
										  int* k_Q,
										  real* QQ,
										  unsigned int sizeQ,
										  int kArray,
										  real om1,
										  unsigned int* neighborX,
										  unsigned int* neighborY,
										  unsigned int* neighborZ,
										  unsigned int size_Mat,
										  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, sizeQ);

   QVelDeviceCompZeroPress27<<< grid.grid, grid.threads >>> (   nx,
														  ny,
														  vx,
														  vy,
														  vz,
														  DD,
														  k_Q,
														  QQ,
														  sizeQ,
														  //numberOfBCnodes,
														  om1,
														  neighborX,
														  neighborY,
														  neighborZ,
														  size_Mat,
														  isEvenTimestep);
   getLastCudaError("QVelDeviceCompZeroPress27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDevIncompHighNu27(unsigned int numberOfThreads,
									  int nx,
									  int ny,
									  real* vx,
									  real* vy,
									  real* vz,
									  real* DD,
									  int* k_Q,
									  real* QQ,
									  unsigned int sizeQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDeviceIncompHighNu27<<< grid.grid, grid.threads >>> (nx,
													  ny,
													  vx,
													  vy,
													  vz,
													  DD,
													  k_Q,
													  QQ,
													  sizeQ,
													  numberOfBCnodes,
													  om1,
													  neighborX,
													  neighborY,
													  neighborZ,
													  size_Mat,
													  isEvenTimestep);
      getLastCudaError("QVelDeviceIncompHighNu27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVelDevCompHighNu27(  unsigned int numberOfThreads,
									  int nx,
									  int ny,
									  real* vx,
									  real* vy,
									  real* vz,
									  real* DD,
									  int* k_Q,
									  real* QQ,
									  unsigned int sizeQ,
									  unsigned int numberOfBCnodes,
									  real om1,
									  unsigned int* neighborX,
									  unsigned int* neighborY,
									  unsigned int* neighborZ,
									  unsigned int size_Mat,
									  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVelDeviceCompHighNu27<<< grid.grid, grid.threads >>> (  nx,
													  ny,
													  vx,
													  vy,
													  vz,
													  DD,
													  k_Q,
													  QQ,
													  sizeQ,
													  numberOfBCnodes,
													  om1,
													  neighborX,
													  neighborY,
													  neighborZ,
													  size_Mat,
													  isEvenTimestep);
      getLastCudaError("QVelDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVeloDevEQ27(unsigned int numberOfThreads,
							 real* VeloX,
							 real* VeloY,
							 real* VeloZ,
							 real* DD,
							 int* k_Q,
							 int numberOfBCnodes,
							 real om1,
							 unsigned int* neighborX,
							 unsigned int* neighborY,
							 unsigned int* neighborZ,
							 unsigned int size_Mat,
							 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QVeloDeviceEQ27<<< grid.grid, grid.threads >>> (VeloX,
											 VeloY,
											 VeloZ,
											 DD,
											 k_Q,
											 numberOfBCnodes,
											 om1,
											 neighborX,
											 neighborY,
											 neighborZ,
											 size_Mat,
											 isEvenTimestep);
      getLastCudaError("QVeloDeviceEQ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QVeloStreetDevEQ27(
	uint  numberOfThreads,
	real* veloXfraction,
	real* veloYfraction,
	int*  naschVelo,
	real* DD,
	int*  naschIndex,
	int   numberOfStreetNodes,
	real  velocityRatio,
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	uint  size_Mat,
	bool  isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfStreetNodes);

	QVeloStreetDeviceEQ27 << < grid.grid, grid.threads >> > (
		veloXfraction,
		veloYfraction,
		naschVelo,
		DD,
		naschIndex,
		numberOfStreetNodes,
		velocityRatio,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("QVeloStreetDeviceEQ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QSlipDev27(unsigned int numberOfThreads,
                           real* DD,
                           int* k_Q,
                           real* QQ,
                           unsigned int sizeQ,
                           real om1,
                           unsigned int* neighborX,
                           unsigned int* neighborY,
                           unsigned int* neighborZ,
                           unsigned int size_Mat,
                           bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, sizeQ);

   QSlipDevice27<<< grid.grid, grid.threads >>> (DD,
                                           k_Q,
                                           QQ,
                                           sizeQ,
                                           om1,
                                           neighborX,
                                           neighborY,
                                           neighborZ,
                                           size_Mat,
                                           isEvenTimestep);
   getLastCudaError("QSlipDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QSlipDevComp27(unsigned int numberOfThreads,
							   real* DD,
							   int* k_Q,
							   real* QQ,
							   unsigned int sizeQ,
							   real om1,
							   unsigned int* neighborX,
							   unsigned int* neighborY,
							   unsigned int* neighborZ,
                        real* turbViscosity,
                        bool useTurbViscosity,
							   unsigned int size_Mat,
							   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, sizeQ);

   if(useTurbViscosity)
   {
      QSlipDeviceComp27TurbViscosity<<< grid.grid, grid.threads >>> (DD,
											   k_Q,
											   QQ,
											   sizeQ,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
                                    turbViscosity,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QSlipDeviceComp27TurbViscosity execution failed");
   }
   else
   {
      QSlipDeviceComp27<<< grid.grid, grid.threads >>> (DD,
											   k_Q,
											   QQ,
											   sizeQ,
											   om1,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
      getLastCudaError("QSlipDeviceComp27 execution failed");
   }
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QSlipGeomDevComp27(unsigned int numberOfThreads,
								   real* DD,
								   int* k_Q,
								   real* QQ,
								   unsigned int sizeQ,
								   real om1,
								   real* NormalX,
								   real* NormalY,
								   real* NormalZ,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, sizeQ);

   QSlipGeomDeviceComp27<<< grid.grid, grid.threads >>> (DD,
												   k_Q,
												   QQ,
												   sizeQ,
												   om1,
												   NormalX,
												   NormalY,
												   NormalZ,
												   neighborX,
												   neighborY,
												   neighborZ,
												   size_Mat,
												   isEvenTimestep);
   getLastCudaError("QSlipGeomDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QSlipNormDevComp27(unsigned int numberOfThreads,
								   real* DD,
								   int* k_Q,
								   real* QQ,
								   unsigned int sizeQ,
								   real om1,
								   real* NormalX,
								   real* NormalY,
								   real* NormalZ,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, sizeQ);

   QSlipNormDeviceComp27<<< grid.grid, grid.threads >>> (DD,
												   k_Q,
												   QQ,
												   sizeQ,
												   om1,
												   NormalX,
												   NormalY,
												   NormalZ,
												   neighborX,
												   neighborY,
												   neighborZ,
												   size_Mat,
												   isEvenTimestep);
      getLastCudaError("QSlipGeomDeviceComp27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QStressDevComp27(unsigned int numberOfThreads,
							   real* DD,
							   int* k_Q,
                        int* k_N,
							   real* QQ,
							   unsigned int sizeQ,
							   real om1,
                        real* turbViscosity,
                        real* vx,
                        real* vy,
                        real* vz,
                        real* normalX,
                        real* normalY,
                        real* normalZ,
                        real* vx_bc,
                        real* vy_bc,
                        real* vz_bc,
                        real* vx1,
                        real* vy1,
                        real* vz1,
                        int* samplingOffset,
                        real* z0,
                        bool  hasWallModelMonitor,
                        real* u_star,
                        real* Fx,
                        real* Fy,
                        real* Fz,
							   unsigned int* neighborX,
							   unsigned int* neighborY,
							   unsigned int* neighborZ,
							   unsigned int size_Mat,
							   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, sizeQ);

   QStressDeviceComp27<<< grid.grid, grid.threads >>> (DD,
											   k_Q,
                                    k_N,
											   QQ,
											   sizeQ,
											   om1,
                                    turbViscosity,
                                    vx,
                                    vy,
                                    vz,
                                    normalX,
                                    normalY,
                                    normalZ,
                                    vx_bc,
                                    vy_bc,
                                    vz_bc,
                                    vx1,
                                    vy1,
                                    vz1,
                                    samplingOffset,
                                    z0,
                                    hasWallModelMonitor,
                                    u_star,
                                    Fx,
                                    Fy,
                                    Fz,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
   getLastCudaError("QSlipDeviceComp27 execution failed");
}

//////////////////////////////////////////////////////////////////////////
extern "C" void BBStressDev27(unsigned int numberOfThreads,
							   real* DD,
							   int* k_Q,
                        int* k_N,
							   real* QQ,
							   unsigned int sizeQ,
                        real* vx,
                        real* vy,
                        real* vz,
                        real* normalX,
                        real* normalY,
                        real* normalZ,
                        real* vx_bc,
                        real* vy_bc,
                        real* vz_bc,
                        real* vx1,
                        real* vy1,
                        real* vz1,
                        int* samplingOffset,
                        real* z0,
                        bool  hasWallModelMonitor,
                        real* u_star,
                        real* Fx,
                        real* Fy,
                        real* Fz,
							   unsigned int* neighborX,
							   unsigned int* neighborY,
							   unsigned int* neighborZ,
							   unsigned int size_Mat,
							   bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, sizeQ);

   BBStressDevice27<<< grid.grid, grid.threads >>> (DD,
											   k_Q,
                                    k_N,
											   QQ,
											   sizeQ,
                                    vx,
                                    vy,
                                    vz,
                                    normalX,
                                    normalY,
                                    normalZ,
                                    vx_bc,
                                    vy_bc,
                                    vz_bc,
                                    vx1,
                                    vy1,
                                    vz1,
                                    samplingOffset,
                                    z0,
                                    hasWallModelMonitor,
                                    u_star,
                                    Fx,
                                    Fy,
                                    Fz,
											   neighborX,
											   neighborY,
											   neighborZ,
											   size_Mat,
											   isEvenTimestep);
   getLastCudaError("BBStressDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDev27(unsigned int numberOfThreads,
                             int nx,
                             int ny,
                             real* rhoBC,
                             real* DD,
                             int* k_Q,
                             real* QQ,
                             unsigned int sizeQ,
                             unsigned int numberOfBCnodes,
                             real om1,
                             unsigned int* neighborX,
                             unsigned int* neighborY,
                             unsigned int* neighborZ,
                             unsigned int size_Mat,
                             bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDevice27<<< grid.grid, grid.threads >>> (nx,
                                             ny,
                                             rhoBC,
                                             DD,
                                             k_Q,
                                             QQ,
                                             sizeQ,
                                             numberOfBCnodes,
                                             om1,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             isEvenTimestep);
   getLastCudaError("QPressDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevAntiBB27(  unsigned int numberOfThreads,
                                    real* rhoBC,
									real* vx,
									real* vy,
									real* vz,
									real* DD,
									int* k_Q,
									real* QQ,
									int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceAntiBB27<<< grid.grid, grid.threads >>>( rhoBC,
												vx,
												vy,
												vz,
												DD,
												k_Q,
												QQ,
												numberOfBCnodes,
												om1,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep);
   getLastCudaError("QPressDeviceAntiBB27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevFixBackflow27( unsigned int numberOfThreads,
                                        real* rhoBC,
                                        real* DD,
                                        int* k_Q,
                                        unsigned int numberOfBCnodes,
                                        real om1,
                                        unsigned int* neighborX,
                                        unsigned int* neighborY,
                                        unsigned int* neighborZ,
                                        unsigned int size_Mat,
                                        bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceFixBackflow27<<< grid.grid, grid.threads >>> (  rhoBC,
                                                         DD,
                                                         k_Q,
                                                         numberOfBCnodes,
                                                         om1,
                                                         neighborX,
                                                         neighborY,
                                                         neighborZ,
                                                         size_Mat,
                                                         isEvenTimestep);
   getLastCudaError("QPressDeviceFixBackflow27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevDirDepBot27(  unsigned int numberOfThreads,
                                       real* rhoBC,
                                       real* DD,
                                       int* k_Q,
                                       unsigned int numberOfBCnodes,
                                       real om1,
                                       unsigned int* neighborX,
                                       unsigned int* neighborY,
                                       unsigned int* neighborZ,
                                       unsigned int size_Mat,
                                       bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceDirDepBot27<<< grid.grid, grid.threads >>> ( rhoBC,
                                                      DD,
                                                      k_Q,
                                                      numberOfBCnodes,
                                                      om1,
                                                      neighborX,
                                                      neighborY,
                                                      neighborZ,
                                                      size_Mat,
                                                      isEvenTimestep);
   getLastCudaError("QPressDeviceDirDepBot27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressNoRhoDev27(unsigned int numberOfThreads,
                                 real* rhoBC,
                                 real* DD,
                                 int* k_Q,
                                 int* k_N,
                                 unsigned int numberOfBCnodes,
                                 real om1,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 unsigned int size_Mat,
                                 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressNoRhoDevice27<<< grid.grid, grid.threads >>> (   rhoBC,
													DD,
													k_Q,
													k_N,
													numberOfBCnodes,
													om1,
													neighborX,
													neighborY,
													neighborZ,
													size_Mat,
													isEvenTimestep);
   getLastCudaError("QPressNoRhoDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QInflowScaleByPressDev27(unsigned int numberOfThreads,
										 real* rhoBC,
										 real* DD,
										 int* k_Q,
										 int* k_N,
										 unsigned int numberOfBCnodes,
										 real om1,
										 unsigned int* neighborX,
										 unsigned int* neighborY,
										 unsigned int* neighborZ,
										 unsigned int size_Mat,
										 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QInflowScaleByPressDevice27<<< grid.grid, grid.threads >>> (  rhoBC,
														DD,
														k_Q,
														k_N,
														numberOfBCnodes,
														om1,
														neighborX,
														neighborY,
														neighborZ,
														size_Mat,
														isEvenTimestep);
   getLastCudaError("QInflowScaleByPressDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevOld27(  unsigned int numberOfThreads,
                                     real* rhoBC,
                                     real* DD,
                                     int* k_Q,
                                     int* k_N,
                                     unsigned int numberOfBCnodes,
                                     real om1,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceOld27<<< grid.grid, grid.threads >>> ( rhoBC,
                                                DD,
                                                k_Q,
                                                k_N,
                                                numberOfBCnodes,
                                                om1,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                isEvenTimestep);
   getLastCudaError("QPressDeviceOld27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevIncompNEQ27(unsigned int numberOfThreads,
                                     real* rhoBC,
                                     real* DD,
                                     int* k_Q,
                                     int* k_N,
                                     unsigned int numberOfBCnodes,
                                     real om1,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceIncompNEQ27<<< grid.grid, grid.threads >>> (   rhoBC,
														DD,
														k_Q,
														k_N,
														numberOfBCnodes,
														om1,
														neighborX,
														neighborY,
														neighborZ,
														size_Mat,
														isEvenTimestep);
   getLastCudaError("QPressDeviceIncompNEQ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevNEQ27(  unsigned int numberOfThreads,
                                     real* rhoBC,
                                     real* DD,
                                     int* k_Q,
                                     int* k_N,
                                     unsigned int numberOfBCnodes,
                                     real om1,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceNEQ27<<< grid.grid, grid.threads >>> ( rhoBC,
                                                DD,
                                                k_Q,
                                                k_N,
                                                numberOfBCnodes,
                                                om1,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                isEvenTimestep);
   getLastCudaError("QPressDeviceOld27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevEQZ27(  unsigned int numberOfThreads,
                                     real* rhoBC,
                                     real* DD,
                                     int* k_Q,
                                     int* k_N,
                                     real* kTestRE,
                                     unsigned int numberOfBCnodes,
                                     real om1,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceEQZ27<<< grid.grid, grid.threads >>> ( rhoBC,
                                                DD,
                                                k_Q,
                                                k_N,
                                                kTestRE,
                                                numberOfBCnodes,
                                                om1,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                isEvenTimestep);
   getLastCudaError("QPressDeviceEQZ27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevZero27(unsigned int numberOfThreads,
                                real* DD,
                                int* k_Q,
                                unsigned int numberOfBCnodes,
                                unsigned int* neighborX,
                                unsigned int* neighborY,
                                unsigned int* neighborZ,
                                unsigned int size_Mat,
                                bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPressDeviceZero27<<< grid.grid, grid.threads >>> (DD,
                                                k_Q,
                                                numberOfBCnodes,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                isEvenTimestep);
   getLastCudaError("QPressDeviceOld27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDevFake27(     unsigned int numberOfThreads,
                                     real* rhoBC,
                                     real* DD,
                                     int* k_Q,
                                     int* k_N,
                                     unsigned int numberOfBCnodes,
                                     real om1,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat,
                                     bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      QPressDeviceFake27<<< grid.grid, grid.threads >>> (rhoBC,
                                                DD,
                                                k_Q,
                                                k_N,
                                                numberOfBCnodes,
                                                om1,
                                                neighborX,
                                                neighborY,
                                                neighborZ,
                                                size_Mat,
                                                isEvenTimestep);
      getLastCudaError("QPressDeviceFake27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void BBDev27( unsigned int numberOfThreads,
                       int nx,
                       int ny,
                       real* DD,
                       int* k_Q,
                       real* QQ,
                       unsigned int sizeQ,
                       unsigned int numberOfBCnodes,
                       real om1,
                       unsigned int* neighborX,
                       unsigned int* neighborY,
                       unsigned int* neighborZ,
                       unsigned int size_Mat,
                       bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      BBDevice27<<< grid.grid, grid.threads >>> (  nx,
                                          ny,
                                          DD,
                                          k_Q,
                                          QQ,
                                          sizeQ,
                                          numberOfBCnodes,
                                          om1,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          isEvenTimestep);
      getLastCudaError("BBDevice27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPressDev27_IntBB(  unsigned int numberOfThreads,
									real* rho,
									real* DD,
									int* k_Q,
									real* QQ,
									unsigned int sizeQ,
									unsigned int numberOfBCnodes,
									real om1,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int size_Mat,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

	QPressDevice27_IntBB<<< grid.grid, grid.threads >>> (rho,
													DD,
													k_Q,
													QQ,
													sizeQ,
													numberOfBCnodes,
													om1,
													neighborX,
													neighborY,
													neighborZ,
													size_Mat,
													isEvenTimestep);
	getLastCudaError("QPressDevice27_IntBB execution failed");
}
// TODO: https://git.rz.tu-bs.de/irmb/VirtualFluids_dev/-/issues/29
//////////////////////////////////////////////////////////////////////////
extern "C" void PressSchlaffer27(unsigned int numberOfThreads,
                                 real* rhoBC,
                                 real* DD,
                                 real* vx0,
                                 real* vy0,
                                 real* vz0,
                                 real* deltaVz0,
                                 int* k_Q,
                                 int* k_N,
                                 int numberOfBCnodes,
                                 real om1,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 unsigned int size_Mat,
                                 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   PressSchlaff27<<< grid.grid, grid.threads >>>(  rhoBC,
                                             DD,
                                             vx0,
                                             vy0,
                                             vz0,
                                             deltaVz0,
                                             k_Q,
                                             k_N,
                                             numberOfBCnodes,
                                             om1,
                                             neighborX,
                                             neighborY,
                                             neighborZ,
                                             size_Mat,
                                             isEvenTimestep);
   getLastCudaError("PressSchlaff27 execution failed");
}
// TODO: https://git.rz.tu-bs.de/irmb/VirtualFluids_dev/-/issues/29
//////////////////////////////////////////////////////////////////////////
extern "C" void VelSchlaffer27(  unsigned int numberOfThreads,
                                 int t,
                                 real* DD,
                                 real* vz0,
                                 real* deltaVz0,
                                 int* k_Q,
                                 int* k_N,
                                 int numberOfBCnodes,
                                 real om1,
                                 unsigned int* neighborX,
                                 unsigned int* neighborY,
                                 unsigned int* neighborZ,
                                 unsigned int size_Mat,
                                 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   VelSchlaff27<<< grid.grid, grid.threads >>>( t,
                                          DD,
                                          vz0,
                                          deltaVz0,
                                          k_Q,
                                          k_N,
                                          numberOfBCnodes,
                                          om1,
                                          neighborX,
                                          neighborY,
                                          neighborZ,
                                          size_Mat,
                                          isEvenTimestep);
      getLastCudaError("VelSchlaff27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void QPrecursorDevCompZeroPress(  uint numberOfThreads, real tRatio,
                                             real* DD, real* QQ, int* k_Q, 
                                             uint sizeQ, uint numberOfBCnodes,
                                             real omega, real velocityRatio,
                                             uint* neighborX, uint* neighborY, uint* neighborZ,
                                             uint* neighborsNT, uint* neighborsNB, uint* neighborsST, uint* neighborsSB,
                                             real* weightsNT, real* weightsNB, real* weightsST, real* weightsSB,
                                             real* vxLast, real* vyLast, real* vzLast,
                                             real* vxCurrent, real* vyCurrent, real* vzCurrent,
                                             real velocityX, real velocityY, real velocityZ, 
                                             unsigned long long size_Mat, bool evenOrOdd)
{

   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

   QPrecursorDeviceCompZeroPress<<< grid.grid, grid.threads >>>(k_Q, numberOfBCnodes, sizeQ, omega, DD, QQ,
                                                               neighborX, neighborY, neighborZ,
                                                               neighborsNT, neighborsNB, neighborsST, neighborsSB,
                                                               weightsNT, weightsNB, weightsST, weightsSB,
                                                               vxLast, vyLast, vzLast,
                                                               vxCurrent, vyCurrent, vzCurrent, 
                                                               velocityX, velocityY, velocityZ, 
                                                               tRatio, velocityRatio, size_Mat, evenOrOdd);
   getLastCudaError("QPrecursorDeviceCompZeroPress execution failed"); 


}
//////////////////////////////////////////////////////////////////////////
extern "C" void PropVelo(   unsigned int numberOfThreads,
                            unsigned int* neighborX,
                            unsigned int* neighborY,
                            unsigned int* neighborZ,
                            real* rho,
                            real* ux,
                            real* uy,
                            real* uz,
                            int* k_Q,
							unsigned int size_Prop,
                            unsigned int size_Mat,
                            unsigned int* bcMatD,
                            real* DD,
                            bool EvenOrOdd)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Prop);

      PropellerBC<<< grid.grid, grid.threads >>>(neighborX,
                                       neighborY,
                                       neighborZ,
                                       rho,
                                       ux,
                                       uy,
                                       uz,
									   k_Q,
									   size_Prop,
                                       size_Mat,
									   bcMatD,
                                       DD,
                                       EvenOrOdd);
      getLastCudaError("PropellerBC execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF27( real* DC,
                        real* DF,
                        unsigned int* neighborCX,
                        unsigned int* neighborCY,
                        unsigned int* neighborCZ,
                        unsigned int* neighborFX,
                        unsigned int* neighborFY,
                        unsigned int* neighborFZ,
                        unsigned int size_MatC,
                        unsigned int size_MatF,
                        bool isEvenTimestep,
                        unsigned int* posCSWB,
                        unsigned int* posFSWB,
                        unsigned int kCF,
                        real omCoarse,
                        real omFine,
                        real nu,
                        unsigned int nxC,
                        unsigned int nyC,
                        unsigned int nxF,
                        unsigned int nyF,
                        unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);
   
      scaleCF27<<< grid.grid, grid.threads >>> ( DC,
                                             DF,
                                             neighborCX,
                                             neighborCY,
                                             neighborCZ,
                                             neighborFX,
                                             neighborFY,
                                             neighborFZ,
                                             size_MatC,
                                             size_MatF,
                                             isEvenTimestep,
                                             posCSWB,
                                             posFSWB,
                                             kCF,
                                             omCoarse,
                                             omFine,
                                             nu,
                                             nxC,
                                             nyC,
                                             nxF,
                                             nyF);
      getLastCudaError("scaleCF27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCFEff27(real* DC,
                             real* DF,
                             unsigned int* neighborCX,
                             unsigned int* neighborCY,
                             unsigned int* neighborCZ,
                             unsigned int* neighborFX,
                             unsigned int* neighborFY,
                             unsigned int* neighborFZ,
                             unsigned int size_MatC,
                             unsigned int size_MatF,
                             bool isEvenTimestep,
                             unsigned int* posCSWB,
                             unsigned int* posFSWB,
                             unsigned int kCF,
                             real omCoarse,
                             real omFine,
                             real nu,
                             unsigned int nxC,
                             unsigned int nyC,
                             unsigned int nxF,
                             unsigned int nyF,
                             unsigned int numberOfThreads,
                             OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFEff27<<< grid.grid, grid.threads >>> ( DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offCF);
      getLastCudaError("scaleCFEff27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCFLast27(real* DC,
                              real* DF,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posCSWB,
                              unsigned int* posFSWB,
                              unsigned int kCF,
                              real omCoarse,
                              real omFine,
                              real nu,
                              unsigned int nxC,
                              unsigned int nyC,
                              unsigned int nxF,
                              unsigned int nyF,
                              unsigned int numberOfThreads,
                              OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFLast27<<< grid.grid, grid.threads >>> (DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offCF);
      getLastCudaError("scaleCFLast27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCFpress27(  real* DC,
                                 real* DF,
                                 unsigned int* neighborCX,
                                 unsigned int* neighborCY,
                                 unsigned int* neighborCZ,
                                 unsigned int* neighborFX,
                                 unsigned int* neighborFY,
                                 unsigned int* neighborFZ,
                                 unsigned int size_MatC,
                                 unsigned int size_MatF,
                                 bool isEvenTimestep,
                                 unsigned int* posCSWB,
                                 unsigned int* posFSWB,
                                 unsigned int kCF,
                                 real omCoarse,
                                 real omFine,
                                 real nu,
                                 unsigned int nxC,
                                 unsigned int nyC,
                                 unsigned int nxF,
                                 unsigned int nyF,
                                 unsigned int numberOfThreads,
                                 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFpress27<<< grid.grid, grid.threads >>>(DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offCF);
      getLastCudaError("scaleCFpress27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_Fix_27(  real* DC,
                                 real* DF,
                                 unsigned int* neighborCX,
                                 unsigned int* neighborCY,
                                 unsigned int* neighborCZ,
                                 unsigned int* neighborFX,
                                 unsigned int* neighborFY,
                                 unsigned int* neighborFZ,
                                 unsigned int size_MatC,
                                 unsigned int size_MatF,
                                 bool isEvenTimestep,
                                 unsigned int* posCSWB,
                                 unsigned int* posFSWB,
                                 unsigned int kCF,
                                 real omCoarse,
                                 real omFine,
                                 real nu,
                                 unsigned int nxC,
                                 unsigned int nyC,
                                 unsigned int nxF,
                                 unsigned int nyF,
                                 unsigned int numberOfThreads,
                                 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_Fix_27<<< grid.grid, grid.threads >>>(DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offCF);
      getLastCudaError("scaleCF_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_Fix_comp_27( real* DC,
									 real* DF,
									 unsigned int* neighborCX,
									 unsigned int* neighborCY,
									 unsigned int* neighborCZ,
									 unsigned int* neighborFX,
									 unsigned int* neighborFY,
									 unsigned int* neighborFZ,
									 unsigned int size_MatC,
									 unsigned int size_MatF,
									 bool isEvenTimestep,
									 unsigned int* posCSWB,
									 unsigned int* posFSWB,
									 unsigned int kCF,
									 real omCoarse,
									 real omFine,
									 real nu,
									 unsigned int nxC,
									 unsigned int nyC,
									 unsigned int nxF,
									 unsigned int nyF,
									 unsigned int numberOfThreads,
									 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_Fix_comp_27<<< grid.grid, grid.threads >>>(   DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_0817_comp_27(real* DC,
									 real* DF,
									 unsigned int* neighborCX,
									 unsigned int* neighborCY,
									 unsigned int* neighborCZ,
									 unsigned int* neighborFX,
									 unsigned int* neighborFY,
									 unsigned int* neighborFZ,
									 unsigned int size_MatC,
									 unsigned int size_MatF,
									 bool isEvenTimestep,
									 unsigned int* posCSWB,
									 unsigned int* posFSWB,
									 unsigned int kCF,
									 real omCoarse,
									 real omFine,
									 real nu,
									 unsigned int nxC,
									 unsigned int nyC,
									 unsigned int nxF,
									 unsigned int nyF,
									 unsigned int numberOfThreads,
									 OffCF offCF,
                            ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_0817_comp_27<<< grid.grid, grid.threads, 0, stream >>>(  DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_0817_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_comp_D3Q27F3_2018(real* DC,
										  real* DF,
										  real* G6,
										  unsigned int* neighborCX,
										  unsigned int* neighborCY,
										  unsigned int* neighborCZ,
										  unsigned int* neighborFX,
										  unsigned int* neighborFY,
										  unsigned int* neighborFZ,
										  unsigned int size_MatC,
										  unsigned int size_MatF,
										  bool isEvenTimestep,
										  unsigned int* posCSWB,
										  unsigned int* posFSWB,
										  unsigned int kCF,
										  real omCoarse,
										  real omFine,
										  real nu,
										  unsigned int nxC,
										  unsigned int nyC,
										  unsigned int nxF,
										  unsigned int nyF,
										  unsigned int numberOfThreads,
										  OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_comp_D3Q27F3_2018 <<< grid.grid, grid.threads >>>(DC,
															DF,
															G6,
															neighborCX,
															neighborCY,
															neighborCZ,
															neighborFX,
															neighborFY,
															neighborFZ,
															size_MatC,
															size_MatF,
															isEvenTimestep,
															posCSWB,
															posFSWB,
															kCF,
															omCoarse,
															omFine,
															nu,
															nxC,
															nyC,
															nxF,
															nyF,
															offCF);
      getLastCudaError("scaleCF_comp_D3Q27F3_2018 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_comp_D3Q27F3(real* DC,
									 real* DF,
									 real* G6,
									 unsigned int* neighborCX,
									 unsigned int* neighborCY,
									 unsigned int* neighborCZ,
									 unsigned int* neighborFX,
									 unsigned int* neighborFY,
									 unsigned int* neighborFZ,
									 unsigned int size_MatC,
									 unsigned int size_MatF,
									 bool isEvenTimestep,
									 unsigned int* posCSWB,
									 unsigned int* posFSWB,
									 unsigned int kCF,
									 real omCoarse,
									 real omFine,
									 real nu,
									 unsigned int nxC,
									 unsigned int nyC,
									 unsigned int nxF,
									 unsigned int nyF,
									 unsigned int numberOfThreads,
									 OffCF offCF,
                            ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_comp_D3Q27F3 <<< grid.grid, grid.threads, 0, stream >>>( DC,
														DF,
														G6,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_comp_D3Q27F3 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_staggered_time_comp_27(  real* DC,
												 real* DF,
												 unsigned int* neighborCX,
												 unsigned int* neighborCY,
												 unsigned int* neighborCZ,
												 unsigned int* neighborFX,
												 unsigned int* neighborFY,
												 unsigned int* neighborFZ,
												 unsigned int size_MatC,
												 unsigned int size_MatF,
												 bool isEvenTimestep,
												 unsigned int* posCSWB,
												 unsigned int* posFSWB,
												 unsigned int kCF,
												 real omCoarse,
												 real omFine,
												 real nu,
												 unsigned int nxC,
												 unsigned int nyC,
												 unsigned int nxF,
												 unsigned int nyF,
												 unsigned int numberOfThreads,
												 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_staggered_time_comp_27<<< grid.grid, grid.threads >>>(    DC,
																	DF,
																	neighborCX,
																	neighborCY,
																	neighborCZ,
																	neighborFX,
																	neighborFY,
																	neighborFZ,
																	size_MatC,
																	size_MatF,
																	isEvenTimestep,
																	posCSWB,
																	posFSWB,
																	kCF,
																	omCoarse,
																	omFine,
																	nu,
																	nxC,
																	nyC,
																	nxF,
																	nyF,
																	offCF);
      getLastCudaError("scaleCF_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_RhoSq_comp_27(   real* DC,
										 real* DF,
										 unsigned int* neighborCX,
										 unsigned int* neighborCY,
										 unsigned int* neighborCZ,
										 unsigned int* neighborFX,
										 unsigned int* neighborFY,
										 unsigned int* neighborFZ,
										 unsigned int size_MatC,
										 unsigned int size_MatF,
										 bool isEvenTimestep,
										 unsigned int* posCSWB,
										 unsigned int* posFSWB,
										 unsigned int kCF,
										 real omCoarse,
										 real omFine,
										 real nu,
										 unsigned int nxC,
										 unsigned int nyC,
										 unsigned int nxF,
										 unsigned int nyF,
										 unsigned int numberOfThreads,
										 OffCF offCF,
                               ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_RhoSq_comp_27<<< grid.grid, grid.threads, 0, stream >>>( DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_RhoSq_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_RhoSq_3rdMom_comp_27(real* DC,
											 real* DF,
											 unsigned int* neighborCX,
											 unsigned int* neighborCY,
											 unsigned int* neighborCZ,
											 unsigned int* neighborFX,
											 unsigned int* neighborFY,
											 unsigned int* neighborFZ,
											 unsigned int size_MatC,
											 unsigned int size_MatF,
											 bool isEvenTimestep,
											 unsigned int* posCSWB,
											 unsigned int* posFSWB,
											 unsigned int kCF,
											 real omCoarse,
											 real omFine,
											 real nu,
											 unsigned int nxC,
											 unsigned int nyC,
											 unsigned int nxF,
											 unsigned int nyF,
											 unsigned int numberOfThreads,
											 OffCF offCF,
                                  ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_RhoSq_3rdMom_comp_27<<< grid.grid, grid.threads, 0, stream >>>(  DC,
																DF,
																neighborCX,
																neighborCY,
																neighborCZ,
																neighborFX,
																neighborFY,
																neighborFZ,
																size_MatC,
																size_MatF,
																isEvenTimestep,
																posCSWB,
																posFSWB,
																kCF,
																omCoarse,
																omFine,
																nu,
																nxC,
																nyC,
																nxF,
																nyF,
																offCF);
      getLastCudaError("scaleCF_RhoSq_3rdMom_comp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_AA2016_comp_27(real* DC,
									   real* DF,
									   unsigned int* neighborCX,
									   unsigned int* neighborCY,
									   unsigned int* neighborCZ,
									   unsigned int* neighborFX,
									   unsigned int* neighborFY,
									   unsigned int* neighborFZ,
									   unsigned int size_MatC,
									   unsigned int size_MatF,
									   bool isEvenTimestep,
									   unsigned int* posCSWB,
									   unsigned int* posFSWB,
									   unsigned int kCF,
									   real omCoarse,
									   real omFine,
									   real nu,
									   unsigned int nxC,
									   unsigned int nyC,
									   unsigned int nxF,
									   unsigned int nyF,
									   unsigned int numberOfThreads,
									   OffCF offCF,
                              ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_AA2016_comp_27<<< grid.grid, grid.threads, 0, stream >>>(DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posCSWB,
														posFSWB,
														kCF,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offCF);
      getLastCudaError("scaleCF_AA2016_comp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCF_NSPress_27(  real* DC,
									 real* DF,
									 unsigned int* neighborCX,
									 unsigned int* neighborCY,
									 unsigned int* neighborCZ,
									 unsigned int* neighborFX,
									 unsigned int* neighborFY,
									 unsigned int* neighborFZ,
									 unsigned int size_MatC,
									 unsigned int size_MatF,
									 bool isEvenTimestep,
									 unsigned int* posCSWB,
									 unsigned int* posFSWB,
									 unsigned int kCF,
									 real omCoarse,
									 real omFine,
									 real nu,
									 unsigned int nxC,
									 unsigned int nyC,
									 unsigned int nxF,
									 unsigned int nyF,
									 unsigned int numberOfThreads,
									 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCF_NSPress_27<<< grid.grid, grid.threads >>>(DC,
													DF,
													neighborCX,
													neighborCY,
													neighborCZ,
													neighborFX,
													neighborFY,
													neighborFZ,
													size_MatC,
													size_MatF,
													isEvenTimestep,
													posCSWB,
													posFSWB,
													kCF,
													omCoarse,
													omFine,
													nu,
													nxC,
													nyC,
													nxF,
													nyF,
													offCF);
      getLastCudaError("scaleCF_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCFThSMG7(   real* DC,
                                 real* DF,
                                 real* DD7C,
                                 real* DD7F,
                                 unsigned int* neighborCX,
                                 unsigned int* neighborCY,
                                 unsigned int* neighborCZ,
                                 unsigned int* neighborFX,
                                 unsigned int* neighborFY,
                                 unsigned int* neighborFZ,
                                 unsigned int size_MatC,
                                 unsigned int size_MatF,
                                 bool isEvenTimestep,
                                 unsigned int* posCSWB,
                                 unsigned int* posFSWB,
                                 unsigned int kCF,
                                 real nu,
                                 real diffusivity_fine,
                                 unsigned int numberOfThreads,
                                 OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFThSMG7<<< grid.grid, grid.threads >>> (DC,
                                                DF,
                                                DD7C,
                                                DD7F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                nu,
                                                diffusivity_fine,
                                                offCF);
      getLastCudaError("scaleCFThSMG7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCFThS7(  real* DC,
                              real* DF,
                              real* DD7C,
                              real* DD7F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posCSWB,
                              unsigned int* posFSWB,
                              unsigned int kCF,
                              real nu,
                              real diffusivity_fine,
                              unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFThS7<<< grid.grid, grid.threads >>> (  DC,
                                                DF,
                                                DD7C,
                                                DD7F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                nu,
                                                diffusivity_fine);
      getLastCudaError("scaleCFThS7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleCFThS27( real* DC,
                              real* DF,
                              real* DD27C,
                              real* DD27F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posCSWB,
                              unsigned int* posFSWB,
                              unsigned int kCF,
                              real nu,
                              real diffusivity_fine,
                              unsigned int numberOfThreads,
							  OffCF offCF)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

      scaleCFThS27<<< grid.grid, grid.threads >>> ( DC,
                                                DF,
                                                DD27C,
                                                DD27F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posCSWB,
                                                posFSWB,
                                                kCF,
                                                nu,
                                                diffusivity_fine,
										        offCF);
      getLastCudaError("scaleCFThS27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC27( real* DC,
                           real* DF,
                           unsigned int* neighborCX,
                           unsigned int* neighborCY,
                           unsigned int* neighborCZ,
                           unsigned int* neighborFX,
                           unsigned int* neighborFY,
                           unsigned int* neighborFZ,
                           unsigned int size_MatC,
                           unsigned int size_MatF,
                           bool isEvenTimestep,
                           unsigned int* posC,
                           unsigned int* posFSWB,
                           unsigned int kFC,
                           real omCoarse,
                           real omFine,
                           real nu,
                           unsigned int nxC,
                           unsigned int nyC,
                           unsigned int nxF,
                           unsigned int nyF,
                           unsigned int numberOfThreads)
{
   
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC27<<< grid.grid, grid.threads >>> ( DC,
                                             DF,
                                             neighborCX,
                                             neighborCY,
                                             neighborCZ,
                                             neighborFX,
                                             neighborFY,
                                             neighborFZ,
                                             size_MatC,
                                             size_MatF,
                                             isEvenTimestep,
                                             posC,
                                             posFSWB,
                                             kFC,
                                             omCoarse,
                                             omFine,
                                             nu,
                                             nxC,
                                             nyC,
                                             nxF,
                                             nyF);
      getLastCudaError("scaleFC27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFCEff27(real* DC,
                             real* DF,
                             unsigned int* neighborCX,
                             unsigned int* neighborCY,
                             unsigned int* neighborCZ,
                             unsigned int* neighborFX,
                             unsigned int* neighborFY,
                             unsigned int* neighborFZ,
                             unsigned int size_MatC,
                             unsigned int size_MatF,
                             bool isEvenTimestep,
                             unsigned int* posC,
                             unsigned int* posFSWB,
                             unsigned int kFC,
                             real omCoarse,
                             real omFine,
                             real nu,
                             unsigned int nxC,
                             unsigned int nyC,
                             unsigned int nxF,
                             unsigned int nyF,
                             unsigned int numberOfThreads,
                             OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCEff27<<< grid.grid, grid.threads >>> ( DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posC,
                                                posFSWB,
                                                kFC,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offFC);
      getLastCudaError("scaleFCEff27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFCLast27(real* DC,
                              real* DF,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real omCoarse,
                              real omFine,
                              real nu,
                              unsigned int nxC,
                              unsigned int nyC,
                              unsigned int nxF,
                              unsigned int nyF,
                              unsigned int numberOfThreads,
                              OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCLast27<<< grid.grid, grid.threads >>> (DC,
                                                DF,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posC,
                                                posFSWB,
                                                kFC,
                                                omCoarse,
                                                omFine,
                                                nu,
                                                nxC,
                                                nyC,
                                                nxF,
                                                nyF,
                                                offFC);
      getLastCudaError("Kernel execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFCpress27(real* DC,
                              real* DF,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real omCoarse,
                              real omFine,
                              real nu,
                              unsigned int nxC,
                              unsigned int nyC,
                              unsigned int nxF,
                              unsigned int nyF,
                              unsigned int numberOfThreads,
                              OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCpress27<<< grid.grid, grid.threads >>> (  DC,
                                                   DF,
                                                   neighborCX,
                                                   neighborCY,
                                                   neighborCZ,
                                                   neighborFX,
                                                   neighborFY,
                                                   neighborFZ,
                                                   size_MatC,
                                                   size_MatF,
                                                   isEvenTimestep,
                                                   posC,
                                                   posFSWB,
                                                   kFC,
                                                   omCoarse,
                                                   omFine,
                                                   nu,
                                                   nxC,
                                                   nyC,
                                                   nxF,
                                                   nyF,
                                                   offFC);
      getLastCudaError("scaleFCpress27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_Fix_27(real* DC,
                              real* DF,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real omCoarse,
                              real omFine,
                              real nu,
                              unsigned int nxC,
                              unsigned int nyC,
                              unsigned int nxF,
                              unsigned int nyF,
                              unsigned int numberOfThreads,
                              OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_Fix_27<<< grid.grid, grid.threads >>> (  DC,
                                                   DF,
                                                   neighborCX,
                                                   neighborCY,
                                                   neighborCZ,
                                                   neighborFX,
                                                   neighborFY,
                                                   neighborFZ,
                                                   size_MatC,
                                                   size_MatF,
                                                   isEvenTimestep,
                                                   posC,
                                                   posFSWB,
                                                   kFC,
                                                   omCoarse,
                                                   omFine,
                                                   nu,
                                                   nxC,
                                                   nyC,
                                                   nxF,
                                                   nyF,
                                                   offFC);
      getLastCudaError("scaleFC_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_Fix_comp_27(  real* DC,
									  real* DF,
									  unsigned int* neighborCX,
									  unsigned int* neighborCY,
									  unsigned int* neighborCZ,
									  unsigned int* neighborFX,
									  unsigned int* neighborFY,
									  unsigned int* neighborFZ,
									  unsigned int size_MatC,
									  unsigned int size_MatF,
									  bool isEvenTimestep,
									  unsigned int* posC,
									  unsigned int* posFSWB,
									  unsigned int kFC,
									  real omCoarse,
									  real omFine,
									  real nu,
									  unsigned int nxC,
									  unsigned int nyC,
									  unsigned int nxF,
									  unsigned int nyF,
									  unsigned int numberOfThreads,
									  OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_Fix_comp_27<<< grid.grid, grid.threads >>> ( DC,
													   DF,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_0817_comp_27( real* DC,
									  real* DF,
									  unsigned int* neighborCX,
									  unsigned int* neighborCY,
									  unsigned int* neighborCZ,
									  unsigned int* neighborFX,
									  unsigned int* neighborFY,
									  unsigned int* neighborFZ,
									  unsigned int size_MatC,
									  unsigned int size_MatF,
									  bool isEvenTimestep,
									  unsigned int* posC,
									  unsigned int* posFSWB,
									  unsigned int kFC,
									  real omCoarse,
									  real omFine,
									  real nu,
									  unsigned int nxC,
									  unsigned int nyC,
									  unsigned int nxF,
									  unsigned int nyF,
									  unsigned int numberOfThreads,
									  OffFC offFC,
                             ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_0817_comp_27<<< grid.grid, grid.threads, 0, stream >>> (DC,
													   DF,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_0817_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_comp_D3Q27F3_2018( real* DC,
										   real* DF,
										   real* G6,
										   unsigned int* neighborCX,
										   unsigned int* neighborCY,
										   unsigned int* neighborCZ,
										   unsigned int* neighborFX,
										   unsigned int* neighborFY,
										   unsigned int* neighborFZ,
										   unsigned int size_MatC,
										   unsigned int size_MatF,
										   bool isEvenTimestep,
										   unsigned int* posC,
										   unsigned int* posFSWB,
										   unsigned int kFC,
										   real omCoarse,
										   real omFine,
										   real nu,
										   unsigned int nxC,
										   unsigned int nyC,
										   unsigned int nxF,
										   unsigned int nyF,
										   unsigned int numberOfThreads,
										   OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

     scaleFC_comp_D3Q27F3_2018 <<< grid.grid, grid.threads >>> (DC,
															DF,
															G6,
															neighborCX,
															neighborCY,
															neighborCZ,
															neighborFX,
															neighborFY,
															neighborFZ,
															size_MatC,
															size_MatF,
															isEvenTimestep,
															posC,
															posFSWB,
															kFC,
															omCoarse,
															omFine,
															nu,
															nxC,
															nyC,
															nxF,
															nyF,
															offFC);
      getLastCudaError("scaleFC_comp_D3Q27F3_2018 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_comp_D3Q27F3( real* DC,
									  real* DF,
									  real* G6,
									  unsigned int* neighborCX,
									  unsigned int* neighborCY,
									  unsigned int* neighborCZ,
									  unsigned int* neighborFX,
									  unsigned int* neighborFY,
									  unsigned int* neighborFZ,
									  unsigned int size_MatC,
									  unsigned int size_MatF,
									  bool isEvenTimestep,
									  unsigned int* posC,
									  unsigned int* posFSWB,
									  unsigned int kFC,
									  real omCoarse,
									  real omFine,
									  real nu,
									  unsigned int nxC,
									  unsigned int nyC,
									  unsigned int nxF,
									  unsigned int nyF,
									  unsigned int numberOfThreads,
									  OffFC offFC,
                             ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

     scaleFC_comp_D3Q27F3 <<< grid.grid, grid.threads, 0, stream >>> (DC,
													   DF,
													   G6,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_0817_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_staggered_time_comp_27(   real* DC,
												  real* DF,
												  unsigned int* neighborCX,
												  unsigned int* neighborCY,
												  unsigned int* neighborCZ,
												  unsigned int* neighborFX,
												  unsigned int* neighborFY,
												  unsigned int* neighborFZ,
												  unsigned int size_MatC,
												  unsigned int size_MatF,
												  bool isEvenTimestep,
												  unsigned int* posC,
												  unsigned int* posFSWB,
												  unsigned int kFC,
												  real omCoarse,
												  real omFine,
												  real nu,
												  unsigned int nxC,
												  unsigned int nyC,
												  unsigned int nxF,
												  unsigned int nyF,
												  unsigned int numberOfThreads,
												  OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_staggered_time_comp_27<<< grid.grid, grid.threads >>> (  DC,
																   DF,
																   neighborCX,
																   neighborCY,
																   neighborCZ,
																   neighborFX,
																   neighborFY,
																   neighborFZ,
																   size_MatC,
																   size_MatF,
																   isEvenTimestep,
																   posC,
																   posFSWB,
																   kFC,
																   omCoarse,
																   omFine,
																   nu,
																   nxC,
																   nyC,
																   nxF,
																   nyF,
																   offFC);
      getLastCudaError("scaleFC_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_RhoSq_comp_27(real* DC,
									  real* DF,
									  unsigned int* neighborCX,
									  unsigned int* neighborCY,
									  unsigned int* neighborCZ,
									  unsigned int* neighborFX,
									  unsigned int* neighborFY,
									  unsigned int* neighborFZ,
									  unsigned int size_MatC,
									  unsigned int size_MatF,
									  bool isEvenTimestep,
									  unsigned int* posC,
									  unsigned int* posFSWB,
									  unsigned int kFC,
									  real omCoarse,
									  real omFine,
									  real nu,
									  unsigned int nxC,
									  unsigned int nyC,
									  unsigned int nxF,
									  unsigned int nyF,
									  unsigned int numberOfThreads,
									  OffFC offFC,
                             ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_RhoSq_comp_27<<<grid.grid, grid.threads, 0, stream>>>(
													   DC,
													   DF,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_RhoSq_27 execution failed");
}

//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_RhoSq_3rdMom_comp_27( real* DC,
											  real* DF,
											  unsigned int* neighborCX,
											  unsigned int* neighborCY,
											  unsigned int* neighborCZ,
											  unsigned int* neighborFX,
											  unsigned int* neighborFY,
											  unsigned int* neighborFZ,
											  unsigned int size_MatC,
											  unsigned int size_MatF,
											  bool isEvenTimestep,
											  unsigned int* posC,
											  unsigned int* posFSWB,
											  unsigned int kFC,
											  real omCoarse,
											  real omFine,
											  real nu,
											  unsigned int nxC,
											  unsigned int nyC,
											  unsigned int nxF,
											  unsigned int nyF,
											  unsigned int numberOfThreads,
											  OffFC offFC,
                                   ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_RhoSq_3rdMom_comp_27<<< grid.grid, grid.threads, 0, stream >>>(DC,
															  DF,
															  neighborCX,
															  neighborCY,
															  neighborCZ,
															  neighborFX,
															  neighborFY,
															  neighborFZ,
															  size_MatC,
															  size_MatF,
															  isEvenTimestep,
															  posC,
															  posFSWB,
															  kFC,
															  omCoarse,
															  omFine,
															  nu,
															  nxC,
															  nyC,
															  nxF,
															  nyF,
															  offFC);
      getLastCudaError("scaleFC_RhoSq_3rdMom_comp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_AA2016_comp_27( real* DC,
										real* DF,
										unsigned int* neighborCX,
										unsigned int* neighborCY,
										unsigned int* neighborCZ,
										unsigned int* neighborFX,
										unsigned int* neighborFY,
										unsigned int* neighborFZ,
										unsigned int size_MatC,
										unsigned int size_MatF,
										bool isEvenTimestep,
										unsigned int* posC,
										unsigned int* posFSWB,
										unsigned int kFC,
										real omCoarse,
										real omFine,
										real nu,
										unsigned int nxC,
										unsigned int nyC,
										unsigned int nxF,
										unsigned int nyF,
										unsigned int numberOfThreads,
										OffFC offFC,
                              ihipStream_t *stream)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_AA2016_comp_27<<< grid.grid, grid.threads, 0, stream >>>(DC,
														DF,
														neighborCX,
														neighborCY,
														neighborCZ,
														neighborFX,
														neighborFY,
														neighborFZ,
														size_MatC,
														size_MatF,
														isEvenTimestep,
														posC,
														posFSWB,
														kFC,
														omCoarse,
														omFine,
														nu,
														nxC,
														nyC,
														nxF,
														nyF,
														offFC);
      getLastCudaError("scaleFC_AA2016_comp_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFC_NSPress_27(real* DC,
								  real* DF,
								  unsigned int* neighborCX,
								  unsigned int* neighborCY,
								  unsigned int* neighborCZ,
								  unsigned int* neighborFX,
								  unsigned int* neighborFY,
								  unsigned int* neighborFZ,
								  unsigned int size_MatC,
								  unsigned int size_MatF,
								  bool isEvenTimestep,
								  unsigned int* posC,
								  unsigned int* posFSWB,
								  unsigned int kFC,
								  real omCoarse,
								  real omFine,
								  real nu,
								  unsigned int nxC,
								  unsigned int nyC,
								  unsigned int nxF,
								  unsigned int nyF,
								  unsigned int numberOfThreads,
								  OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFC_NSPress_27<<< grid.grid, grid.threads >>> (  DC,
													   DF,
													   neighborCX,
													   neighborCY,
													   neighborCZ,
													   neighborFX,
													   neighborFY,
													   neighborFZ,
													   size_MatC,
													   size_MatF,
													   isEvenTimestep,
													   posC,
													   posFSWB,
													   kFC,
													   omCoarse,
													   omFine,
													   nu,
													   nxC,
													   nyC,
													   nxF,
													   nyF,
													   offFC);
      getLastCudaError("scaleFC_Fix_27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFCThSMG7(real* DC,
                              real* DF,
                              real* DD7C,
                              real* DD7F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real nu,
                              real diffusivity_coarse,
                              unsigned int numberOfThreads,
                              OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCThSMG7<<< grid.grid, grid.threads >>>( DC,
                                                DF,
                                                DD7C,
                                                DD7F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posC,
                                                posFSWB,
                                                kFC,
                                                nu,
                                                diffusivity_coarse,
                                                offFC);
      getLastCudaError("scaleFCThSMG7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFCThS7(  real* DC,
                              real* DF,
                              real* DD7C,
                              real* DD7F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real nu,
                              real diffusivity_coarse,
                              unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCThS7<<< grid.grid, grid.threads >>>(DC,
                                             DF,
                                             DD7C,
                                             DD7F,
                                             neighborCX,
                                             neighborCY,
                                             neighborCZ,
                                             neighborFX,
                                             neighborFY,
                                             neighborFZ,
                                             size_MatC,
                                             size_MatF,
                                             isEvenTimestep,
                                             posC,
                                             posFSWB,
                                             kFC,
                                             nu,
                                             diffusivity_coarse);
      getLastCudaError("scaleFCThS7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void ScaleFCThS27( real* DC,
                              real* DF,
                              real* DD27C,
                              real* DD27F,
                              unsigned int* neighborCX,
                              unsigned int* neighborCY,
                              unsigned int* neighborCZ,
                              unsigned int* neighborFX,
                              unsigned int* neighborFY,
                              unsigned int* neighborFZ,
                              unsigned int size_MatC,
                              unsigned int size_MatF,
                              bool isEvenTimestep,
                              unsigned int* posC,
                              unsigned int* posFSWB,
                              unsigned int kFC,
                              real nu,
                              real diffusivity_coarse,
                              unsigned int numberOfThreads,
							  OffFC offFC)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

      scaleFCThS27<<< grid.grid, grid.threads >>>(  DC,
                                                DF,
                                                DD27C,
                                                DD27F,
                                                neighborCX,
                                                neighborCY,
                                                neighborCZ,
                                                neighborFX,
                                                neighborFY,
                                                neighborFZ,
                                                size_MatC,
                                                size_MatF,
                                                isEvenTimestep,
                                                posC,
                                                posFSWB,
                                                kFC,
                                                nu,
                                                diffusivity_coarse,
												offFC);
      getLastCudaError("scaleFCThS27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void DragLiftPostD27(real* DD,
								int* k_Q,
								real* QQ,
								int numberOfBCnodes,
								double *DragX,
								double *DragY,
								double *DragZ,
								unsigned int* neighborX,
								unsigned int* neighborY,
								unsigned int* neighborZ,
								unsigned int size_Mat,
								bool isEvenTimestep,
								unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

	DragLiftPost27<<< grid.grid, grid.threads >>>(DD,
										k_Q,
										QQ,
										numberOfBCnodes,
										DragX,
										DragY,
										DragZ,
										neighborX,
										neighborY,
										neighborZ,
										size_Mat,
										isEvenTimestep);
	getLastCudaError("DragLift27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void DragLiftPreD27( real* DD,
								int* k_Q,
								real* QQ,
								int numberOfBCnodes,
								double *DragX,
								double *DragY,
								double *DragZ,
								unsigned int* neighborX,
								unsigned int* neighborY,
								unsigned int* neighborZ,
								unsigned int size_Mat,
								bool isEvenTimestep,
								unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);

	DragLiftPre27<<< grid.grid, grid.threads >>>( DD,
										k_Q,
										QQ,
										numberOfBCnodes,
										DragX,
										DragY,
										DragZ,
										neighborX,
										neighborY,
										neighborZ,
										size_Mat,
										isEvenTimestep);
	getLastCudaError("DragLift27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcCPtop27(real* DD,
							int* cpIndex,
							int nonCp,
							double *cpPress,
							unsigned int* neighborX,
							unsigned int* neighborY,
							unsigned int* neighborZ,
							unsigned int size_Mat,
							bool isEvenTimestep,
							unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, nonCp);

	CalcCP27<<< grid.grid, grid.threads >>>(DD,
								  cpIndex,
								  nonCp,
								  cpPress,
								  neighborX,
								  neighborY,
								  neighborZ,
								  size_Mat,
								  isEvenTimestep);
	getLastCudaError("CalcCP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcCPbottom27( real* DD,
								int* cpIndex,
								int nonCp,
								double *cpPress,
								unsigned int* neighborX,
								unsigned int* neighborY,
								unsigned int* neighborZ,
								unsigned int size_Mat,
								bool isEvenTimestep,
								unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, nonCp);

	CalcCP27<<< grid.grid, grid.threads >>>(DD,
								  cpIndex,
								  nonCp,
								  cpPress,
								  neighborX,
								  neighborY,
								  neighborZ,
								  size_Mat,
								  isEvenTimestep);
	getLastCudaError("CalcCP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void GetSendFsPreDev27(real* DD,
								  real* bufferFs,
								  int* sendIndex,
								  int buffmax,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  unsigned int size_Mat,
								  bool isEvenTimestep,
								  unsigned int numberOfThreads,
								  hipStream_t stream)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	getSendFsPre27<<< grid.grid, grid.threads, 0, stream >>>(DD,
										bufferFs,
										sendIndex,
										buffmax,
										neighborX,
										neighborY,
										neighborZ,
										size_Mat,
										isEvenTimestep);
	getLastCudaError("getSendFsPre27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void GetSendFsPostDev27(real* DD,
								   real* bufferFs,
								   int* sendIndex,
								   int buffmax,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep,
								   unsigned int numberOfThreads,
								   hipStream_t stream)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	getSendFsPost27<<< grid.grid, grid.threads, 0, stream >>>(DD,
										 bufferFs,
										 sendIndex,
										 buffmax,
										 neighborX,
										 neighborY,
										 neighborZ,
										 size_Mat,
										 isEvenTimestep);
	getLastCudaError("getSendFsPost27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void SetRecvFsPreDev27(real* DD,
								  real* bufferFs,
								  int* recvIndex,
								  int buffmax,
								  unsigned int* neighborX,
								  unsigned int* neighborY,
								  unsigned int* neighborZ,
								  unsigned int size_Mat,
								  bool isEvenTimestep,
								  unsigned int numberOfThreads,
	                              hipStream_t stream)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	setRecvFsPre27<<< grid.grid, grid.threads, 0, stream >>>(DD,
										bufferFs,
										recvIndex,
										buffmax,
										neighborX,
										neighborY,
										neighborZ,
										size_Mat,
										isEvenTimestep);
	getLastCudaError("setRecvFsPre27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void SetRecvFsPostDev27(real* DD,
								   real* bufferFs,
								   int* recvIndex,
								   int buffmax,
								   unsigned int* neighborX,
								   unsigned int* neighborY,
								   unsigned int* neighborZ,
								   unsigned int size_Mat,
								   bool isEvenTimestep,
	                               unsigned int numberOfThreads,
	                               hipStream_t stream)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	setRecvFsPost27<<< grid.grid, grid.threads, 0, stream >>>(DD,
										 bufferFs,
										 recvIndex,
										 buffmax,
										 neighborX,
										 neighborY,
										 neighborZ,
										 size_Mat,
										 isEvenTimestep);
	getLastCudaError("setRecvFsPost27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void getSendGsDevF3(
	real* G6,
	real* bufferGs,
	int* sendIndex,
	int buffmax,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	bool isEvenTimestep,
	unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	getSendGsF3 <<< grid.grid, grid.threads >>> (
		G6,
		bufferGs,
		sendIndex,
		buffmax,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("getSendGsF3 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void setRecvGsDevF3(
	real* G6,
	real* bufferGs,
	int* recvIndex,
	int buffmax,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	bool isEvenTimestep,
	unsigned int numberOfThreads)
{
	vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, buffmax);

	setRecvGsF3 <<< grid.grid, grid.threads >>> (
		G6,
		bufferGs,
		recvIndex,
		buffmax,
		neighborX,
		neighborY,
		neighborZ,
		size_Mat,
		isEvenTimestep);
	getLastCudaError("setRecvGsF3 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void WallFuncDev27(unsigned int numberOfThreads,
							  int nx,
							  int ny,
							  real* vx,
							  real* vy,
							  real* vz,
							  real* DD,
							  int* k_Q,
							  real* QQ,
							  unsigned int sizeQ,
							  unsigned int numberOfBCnodes,
							  real om1,
							  unsigned int* neighborX,
							  unsigned int* neighborY,
							  unsigned int* neighborZ,
							  unsigned int size_Mat,
							  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfBCnodes);


      WallFunction27<<< grid.grid, grid.threads >>> (  nx,
											  ny,
											  vx,
											  vy,
											  vz,
											  DD,
											  k_Q,
											  QQ,
											  sizeQ,
											  numberOfBCnodes,
											  om1,
											  neighborX,
											  neighborY,
											  neighborZ,
											  size_Mat,
											  isEvenTimestep);
      getLastCudaError("WallFunction27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void SetOutputWallVelocitySP27(unsigned int numberOfThreads,
										  real* vxD,
										  real* vyD,
										  real* vzD,
										  real* vxWall,
										  real* vyWall,
										  real* vzWall,
										  int numberOfWallNodes,
										  int* kWallNodes,
										  real* rhoD,
										  real* pressD,
										  unsigned int* geoD,
										  unsigned int* neighborX,
										  unsigned int* neighborY,
										  unsigned int* neighborZ,
										  unsigned int size_Mat,
										  real* DD,
										  bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfWallNodes);

      LBSetOutputWallVelocitySP27<<< grid.grid, grid.threads >>> (	vxD,
															vyD,
															vzD,
															vxWall,
															vyWall,
															vzWall,
															numberOfWallNodes,
															kWallNodes,
															rhoD,
															pressD,
															geoD,
															neighborX,
															neighborY,
															neighborZ,
															size_Mat,
															DD,
															isEvenTimestep);
      getLastCudaError("LBSetOutputWallVelocitySP27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void GetVelotoForce27(unsigned int numberOfThreads,
								 real* DD,
								 int* bcIndex,
								 int nonAtBC,
								 real* Vx,
								 real* Vy,
								 real* Vz,
								 unsigned int* neighborX,
								 unsigned int* neighborY,
								 unsigned int* neighborZ,
								 unsigned int size_Mat,
								 bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, nonAtBC);

      GetVeloforForcing27<<< grid.grid, grid.threads >>> (DD,
												bcIndex,
												nonAtBC,
												Vx,
												Vy,
												Vz,
												neighborX,
												neighborY,
												neighborZ,
												size_Mat,
												isEvenTimestep);
      getLastCudaError("GetVeloforForcing27 execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void InitParticlesDevice(real* coordX,
									real* coordY,
									real* coordZ,
									real* coordParticleXlocal,
									real* coordParticleYlocal,
									real* coordParticleZlocal,
									real* coordParticleXglobal,
									real* coordParticleYglobal,
									real* coordParticleZglobal,
									real* veloParticleX,
									real* veloParticleY,
									real* veloParticleZ,
									real* randArray,
									unsigned int* particleID,
									unsigned int* cellBaseID,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int* neighborWSB,
							        int level,
									unsigned int numberOfParticles,
									unsigned int size_Mat,
									unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfParticles);

   InitParticles<<< grid.grid, grid.threads >>> (coordX,
										coordY,
										coordZ,
										coordParticleXlocal,
										coordParticleYlocal,
										coordParticleZlocal,
										coordParticleXglobal,
										coordParticleYglobal,
										coordParticleZglobal,
										veloParticleX,
										veloParticleY,
										veloParticleZ,
										randArray,
										particleID,
										cellBaseID,
										bcMatD,
										neighborX,
										neighborY,
										neighborZ,
										neighborWSB,
										level,
										numberOfParticles,
										size_Mat);
      getLastCudaError("InitParticles execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void MoveParticlesDevice(real* coordX,
									real* coordY,
									real* coordZ,
									real* coordParticleXlocal,
									real* coordParticleYlocal,
									real* coordParticleZlocal,
									real* coordParticleXglobal,
									real* coordParticleYglobal,
									real* coordParticleZglobal,
									real* veloParticleX,
									real* veloParticleY,
									real* veloParticleZ,
									real* DD,
									real  omega,
									unsigned int* particleID,
									unsigned int* cellBaseID,
									unsigned int* bcMatD,
									unsigned int* neighborX,
									unsigned int* neighborY,
									unsigned int* neighborZ,
									unsigned int* neighborWSB,
							        int level,
									unsigned int timestep,
									unsigned int numberOfTimesteps,
									unsigned int numberOfParticles,
									unsigned int size_Mat,
									unsigned int numberOfThreads,
									bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfParticles);

   MoveParticles<<< grid.grid, grid.threads >>> (coordX,
										coordY,
										coordZ,
										coordParticleXlocal,
										coordParticleYlocal,
										coordParticleZlocal,
										coordParticleXglobal,
										coordParticleYglobal,
										coordParticleZglobal,
										veloParticleX,
										veloParticleY,
										veloParticleZ,
										DD,
										omega,
										particleID,
										cellBaseID,
										bcMatD,
										neighborX,
										neighborY,
										neighborZ,
										neighborWSB,
										level,
										timestep,
										numberOfTimesteps,
										numberOfParticles,
										size_Mat,
										isEvenTimestep);
      getLastCudaError("MoveParticles execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void initRandomDevice(hiprandState* state,
								 unsigned int size_Mat,
								 unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);
   initRandom<<< grid.grid, grid.threads >>> (state);
   getLastCudaError("initRandom execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void generateRandomValuesDevice( hiprandState* state,
											unsigned int size_Mat,
											real* randArray,
											unsigned int numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);
   generateRandomValues<<< grid.grid, grid.threads >>> (state,randArray);
   getLastCudaError("generateRandomValues execution failed");
}
//////////////////////////////////////////////////////////////////////////
extern "C" void CalcTurbulenceIntensityDevice(
   real* vxx,
   real* vyy,
   real* vzz,
   real* vxy,
   real* vxz,
   real* vyz,
   real* vx_mean,
   real* vy_mean,
   real* vz_mean,
   real* DD,
   uint* typeOfGridNode,
   unsigned int* neighborX,
   unsigned int* neighborY,
   unsigned int* neighborZ,
   unsigned int size_Mat,
   bool isEvenTimestep,
   uint numberOfThreads)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, size_Mat);
   CalcTurbulenceIntensity<<<grid.grid, grid.threads>>>(
     vxx,
     vyy,
     vzz,
	 vxy,
     vxz,
     vyz,
     vx_mean,
     vy_mean,
     vz_mean,
     DD,
     typeOfGridNode,
     neighborX,
     neighborY,
     neighborZ,
     size_Mat,
     isEvenTimestep);

   getLastCudaError("CalcTurbulenceIntensity execution failed");
}













