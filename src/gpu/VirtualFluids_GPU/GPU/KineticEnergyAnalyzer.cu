#include "hip/hip_runtime.h"
#include "KineticEnergyAnalyzer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cmath>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>

#include <iomanip>

#include "cuda/CudaGrid.h"

#include "Parameter/Parameter.h"
// includes, kernels
#include "GPU/GPU_Kernels.cuh"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;

__global__                 void kineticEnergyKernel  (real* vx, real* vy, real* vz, real* rho, uint* neighborX, uint* neighborY, uint* neighborZ, uint* neighborWSB, uint* geo, real* kineticEnergy, uint* isFluid, unsigned long long numberOfLBnodes);

__host__ __device__ inline void kineticEnergyFunction(real* vx, real* vy, real* vz, real* rho, uint* neighborX, uint* neighborY, uint* neighborZ, uint* neighborWSB, uint* geo, real* kineticEnergy, uint* isFluid, uint index);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

bool KineticEnergyAnalyzer::run(uint iter)
{
    if( iter % this->analyzeIter != 0 ) return false;

	int lev = 0;
    vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(para->getParD(lev)->numberofthreads, para->getParD(lev)->numberOfNodes);

    thrust::device_vector<real> kineticEnergy( this->para->getParD(lev)->numberOfNodes, c0o1);
    thrust::device_vector<uint> isFluid      ( this->para->getParD(lev)->numberOfNodes, 0);

    LBCalcMacCompSP27<<< grid.grid, grid.threads >>>(
        para->getParD(lev)->velocityX,
        para->getParD(lev)->velocityY,
        para->getParD(lev)->velocityZ,
        para->getParD(lev)->rho,
        para->getParD(lev)->pressure,
        para->getParD(lev)->typeOfGridNode,
        para->getParD(lev)->neighborX,
        para->getParD(lev)->neighborY,
        para->getParD(lev)->neighborZ,
        para->getParD(lev)->numberOfNodes,
        para->getParD(lev)->distributions.f[0],
        para->getParD(lev)->isEvenTimestep); 
    getLastCudaError("LBCalcMacCompSP27 execution failed"); 

    kineticEnergyKernel<<< grid.grid, grid.threads >>>(
        para->getParD(lev)->velocityX, 
        para->getParD(lev)->velocityY, 
        para->getParD(lev)->velocityZ, 
        para->getParD(lev)->rho, 
        para->getParD(lev)->neighborX,
        para->getParD(lev)->neighborY,
        para->getParD(lev)->neighborZ,
        para->getParD(lev)->neighborInverse,
        para->getParD(lev)->typeOfGridNode,
        kineticEnergy.data().get(), 
        isFluid.data().get(),
        para->getParD(lev)->numberOfNodes);
    hipDeviceSynchronize();

    getLastCudaError("kineticEnergyKernel execution failed");

	 real EKin               = thrust::reduce(kineticEnergy.begin(), kineticEnergy.end(), c0o1, thrust::plus<real>());
     uint numberOfFluidNodes = thrust::reduce(isFluid.begin(),       isFluid.end(),       0,    thrust::plus<uint>());

    //std::cout << "EKin " << EKin << "   " << numberOfFluidNodes << std::endl;

    this->kineticEnergyTimeSeries.push_back( EKin / real(numberOfFluidNodes) );

    //TODO: Should this function probably return nothing?
    return true;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void kineticEnergyKernel(real* vx, real* vy, real* vz, real* rho, uint* neighborX, uint* neighborY, uint* neighborZ, uint* neighborWSB, uint* geo, real* kineticEnergy, uint* isFluid, unsigned long long numberOfLBnodes)
{
    //////////////////////////////////////////////////////////////////////////
    const uint x = threadIdx.x;  // Globaler x-Index 
    const uint y = blockIdx.x;   // Globaler y-Index 
    const uint z = blockIdx.y;   // Globaler z-Index 

    const uint nx = blockDim.x;
    const uint ny = gridDim.x;

    const uint index = nx*(ny*z + y) + x;
	////////////////////////////////////////////////////////////////////////////////
    //printf("%d\n", index);

    //if( index % 34 == 0 || index % 34 == 33 ) return;

    if( index >= (uint)numberOfLBnodes) return;

	unsigned int BC;
	BC = geo[index];
	if (BC != GEO_FLUID) return;

    kineticEnergyFunction( vx, vy, vz, rho, neighborX, neighborY, neighborZ, neighborWSB, geo, kineticEnergy, isFluid, index );
}

__host__ __device__ void kineticEnergyFunction(real* vx, real* vy, real* vz, real* rho, uint* neighborX, uint* neighborY, uint* neighborZ, uint* neighborWSB, uint* geo, real* kineticEnergy, uint* isFluid, uint index)
{
    //////////////////////////////////////////////////////////////////////////////
	//neighbor index                                
	uint k     = index;                             
	uint kPx   = neighborX[k];                      if( geo[ kPx   ] != GEO_FLUID ) return;
	uint kPy   = neighborY[k];                      if( geo[ kPy   ] != GEO_FLUID ) return;
	uint kPz   = neighborZ[k];                      if( geo[ kPz   ] != GEO_FLUID ) return;
	uint kMxyz = neighborWSB[k];                    if( geo[ kMxyz ] != GEO_FLUID ) return;
	uint kMx   = neighborZ[neighborY[kMxyz]];       if( geo[ kMx   ] != GEO_FLUID ) return;
	uint kMy   = neighborZ[neighborX[kMxyz]];       if( geo[ kMy   ] != GEO_FLUID ) return;
	uint kMz   = neighborY[neighborX[kMxyz]];       if( geo[ kMz   ] != GEO_FLUID ) return;
    //////////////////////////////////////////////////////////////////////////

    isFluid[ index ] = 1;

    kineticEnergy[ index ] = c1o2 * ( vx[index] * vx[index] + vy[index] * vy[index] + vz[index] * vz[index] ) * (rho[index] + c1o1);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

KineticEnergyAnalyzer::KineticEnergyAnalyzer(SPtr<Parameter> para, uint analyzeIter)
{
    this->para = para;
    this->analyzeIter = analyzeIter;
}

void KineticEnergyAnalyzer::writeToFile(std::string filename)
{
	std::cout << "KineticEnergyAnalyzer::writeToFile( " << filename << " )" << "\n";

    std::ofstream file;

    file.open(filename + "_KineticEnergyData.dat" );

    for( auto& EKin : this->kineticEnergyTimeSeries )
        file << std::setprecision(15) << EKin << std::endl;

    file.close();

	std::cout << "done!\n";
}


