#include "hip/hip_runtime.h"
//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ / 
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>
#include "KernelUtilities.h"

using namespace vf::lbm::constant;

//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDevice3rdMomentsComp27(
													 real* distributions, 
													 int* subgridDistanceIndices, 
													 real* subgridDistances,
													 unsigned int numberOfBCnodes, 
													 real omega, 
													 unsigned int* neighborX,
													 unsigned int* neighborY,
													 unsigned int* neighborZ,
													 unsigned int numberOfLBnodes, 
													 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &distributions[dirE   *numberOfLBnodes];
      D.f[dirW   ] = &distributions[dirW   *numberOfLBnodes];
      D.f[dirN   ] = &distributions[dirN   *numberOfLBnodes];
      D.f[dirS   ] = &distributions[dirS   *numberOfLBnodes];
      D.f[dirT   ] = &distributions[dirT   *numberOfLBnodes];
      D.f[dirB   ] = &distributions[dirB   *numberOfLBnodes];
      D.f[dirNE  ] = &distributions[dirNE  *numberOfLBnodes];
      D.f[dirSW  ] = &distributions[dirSW  *numberOfLBnodes];
      D.f[dirSE  ] = &distributions[dirSE  *numberOfLBnodes];
      D.f[dirNW  ] = &distributions[dirNW  *numberOfLBnodes];
      D.f[dirTE  ] = &distributions[dirTE  *numberOfLBnodes];
      D.f[dirBW  ] = &distributions[dirBW  *numberOfLBnodes];
      D.f[dirBE  ] = &distributions[dirBE  *numberOfLBnodes];
      D.f[dirTW  ] = &distributions[dirTW  *numberOfLBnodes];
      D.f[dirTN  ] = &distributions[dirTN  *numberOfLBnodes];
      D.f[dirBS  ] = &distributions[dirBS  *numberOfLBnodes];
      D.f[dirBN  ] = &distributions[dirBN  *numberOfLBnodes];
      D.f[dirTS  ] = &distributions[dirTS  *numberOfLBnodes];
      D.f[dirREST] = &distributions[dirREST*numberOfLBnodes];
      D.f[dirTNE ] = &distributions[dirTNE *numberOfLBnodes];
      D.f[dirTSW ] = &distributions[dirTSW *numberOfLBnodes];
      D.f[dirTSE ] = &distributions[dirTSE *numberOfLBnodes];
      D.f[dirTNW ] = &distributions[dirTNW *numberOfLBnodes];
      D.f[dirBNE ] = &distributions[dirBNE *numberOfLBnodes];
      D.f[dirBSW ] = &distributions[dirBSW *numberOfLBnodes];
      D.f[dirBSE ] = &distributions[dirBSE *numberOfLBnodes];
      D.f[dirBNW ] = &distributions[dirBNW *numberOfLBnodes];
   } 
   else
   {
      D.f[dirW   ] = &distributions[dirE   *numberOfLBnodes];
      D.f[dirE   ] = &distributions[dirW   *numberOfLBnodes];
      D.f[dirS   ] = &distributions[dirN   *numberOfLBnodes];
      D.f[dirN   ] = &distributions[dirS   *numberOfLBnodes];
      D.f[dirB   ] = &distributions[dirT   *numberOfLBnodes];
      D.f[dirT   ] = &distributions[dirB   *numberOfLBnodes];
      D.f[dirSW  ] = &distributions[dirNE  *numberOfLBnodes];
      D.f[dirNE  ] = &distributions[dirSW  *numberOfLBnodes];
      D.f[dirNW  ] = &distributions[dirSE  *numberOfLBnodes];
      D.f[dirSE  ] = &distributions[dirNW  *numberOfLBnodes];
      D.f[dirBW  ] = &distributions[dirTE  *numberOfLBnodes];
      D.f[dirTE  ] = &distributions[dirBW  *numberOfLBnodes];
      D.f[dirTW  ] = &distributions[dirBE  *numberOfLBnodes];
      D.f[dirBE  ] = &distributions[dirTW  *numberOfLBnodes];
      D.f[dirBS  ] = &distributions[dirTN  *numberOfLBnodes];
      D.f[dirTN  ] = &distributions[dirBS  *numberOfLBnodes];
      D.f[dirTS  ] = &distributions[dirBN  *numberOfLBnodes];
      D.f[dirBN  ] = &distributions[dirTS  *numberOfLBnodes];
      D.f[dirREST] = &distributions[dirREST*numberOfLBnodes];
      D.f[dirTNE ] = &distributions[dirBSW *numberOfLBnodes];
      D.f[dirTSW ] = &distributions[dirBNE *numberOfLBnodes];
      D.f[dirTSE ] = &distributions[dirBNW *numberOfLBnodes];
      D.f[dirTNW ] = &distributions[dirBSE *numberOfLBnodes];
      D.f[dirBNE ] = &distributions[dirTSW *numberOfLBnodes];
      D.f[dirBSW ] = &distributions[dirTNE *numberOfLBnodes];
      D.f[dirBSE ] = &distributions[dirTNW *numberOfLBnodes];
      D.f[dirBNW ] = &distributions[dirTSE *numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k < numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &subgridDistances[dirE   * numberOfBCnodes];
      q_dirW   = &subgridDistances[dirW   * numberOfBCnodes];
      q_dirN   = &subgridDistances[dirN   * numberOfBCnodes];
      q_dirS   = &subgridDistances[dirS   * numberOfBCnodes];
      q_dirT   = &subgridDistances[dirT   * numberOfBCnodes];
      q_dirB   = &subgridDistances[dirB   * numberOfBCnodes];
      q_dirNE  = &subgridDistances[dirNE  * numberOfBCnodes];
      q_dirSW  = &subgridDistances[dirSW  * numberOfBCnodes];
      q_dirSE  = &subgridDistances[dirSE  * numberOfBCnodes];
      q_dirNW  = &subgridDistances[dirNW  * numberOfBCnodes];
      q_dirTE  = &subgridDistances[dirTE  * numberOfBCnodes];
      q_dirBW  = &subgridDistances[dirBW  * numberOfBCnodes];
      q_dirBE  = &subgridDistances[dirBE  * numberOfBCnodes];
      q_dirTW  = &subgridDistances[dirTW  * numberOfBCnodes];
      q_dirTN  = &subgridDistances[dirTN  * numberOfBCnodes];
      q_dirBS  = &subgridDistances[dirBS  * numberOfBCnodes];
      q_dirBN  = &subgridDistances[dirBN  * numberOfBCnodes];
      q_dirTS  = &subgridDistances[dirTS  * numberOfBCnodes];
      q_dirTNE = &subgridDistances[dirTNE * numberOfBCnodes];
      q_dirTSW = &subgridDistances[dirTSW * numberOfBCnodes];
      q_dirTSE = &subgridDistances[dirTSE * numberOfBCnodes];
      q_dirTNW = &subgridDistances[dirTNW * numberOfBCnodes];
      q_dirBNE = &subgridDistances[dirBNE * numberOfBCnodes];
      q_dirBSW = &subgridDistances[dirBSW * numberOfBCnodes];
      q_dirBSE = &subgridDistances[dirBSE * numberOfBCnodes];
      q_dirBNW = &subgridDistances[dirBNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = subgridDistanceIndices[k];
      unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
            f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dirE   ])[ke   ];
      f_E    = (D.f[dirW   ])[kw   ];
      f_S    = (D.f[dirN   ])[kn   ];
      f_N    = (D.f[dirS   ])[ks   ];
      f_B    = (D.f[dirT   ])[kt   ];
      f_T    = (D.f[dirB   ])[kb   ];
      f_SW   = (D.f[dirNE  ])[kne  ];
      f_NE   = (D.f[dirSW  ])[ksw  ];
      f_NW   = (D.f[dirSE  ])[kse  ];
      f_SE   = (D.f[dirNW  ])[knw  ];
      f_BW   = (D.f[dirTE  ])[kte  ];
      f_TE   = (D.f[dirBW  ])[kbw  ];
      f_TW   = (D.f[dirBE  ])[kbe  ];
      f_BE   = (D.f[dirTW  ])[ktw  ];
      f_BS   = (D.f[dirTN  ])[ktn  ];
      f_TN   = (D.f[dirBS  ])[kbs  ];
      f_TS   = (D.f[dirBN  ])[kbn  ];
      f_BN   = (D.f[dirTS  ])[kts  ];
      f_BSW  = (D.f[dirTNE ])[ktne ];
      f_BNE  = (D.f[dirTSW ])[ktsw ];
      f_BNW  = (D.f[dirTSE ])[ktse ];
      f_BSE  = (D.f[dirTNW ])[ktnw ];
      f_TSW  = (D.f[dirBNE ])[kbne ];
      f_TNE  = (D.f[dirBSW ])[kbsw ];
      f_TNW  = (D.f[dirBSE ])[kbse ];
      f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q, m3;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
				f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
				f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirREST])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE))  + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                  (f_N - f_S)) / (c1o1 + drho); 

      vx3    =    (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                  (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &distributions[dirE   *numberOfLBnodes];
         D.f[dirW   ] = &distributions[dirW   *numberOfLBnodes];
         D.f[dirN   ] = &distributions[dirN   *numberOfLBnodes];
         D.f[dirS   ] = &distributions[dirS   *numberOfLBnodes];
         D.f[dirT   ] = &distributions[dirT   *numberOfLBnodes];
         D.f[dirB   ] = &distributions[dirB   *numberOfLBnodes];
         D.f[dirNE  ] = &distributions[dirNE  *numberOfLBnodes];
         D.f[dirSW  ] = &distributions[dirSW  *numberOfLBnodes];
         D.f[dirSE  ] = &distributions[dirSE  *numberOfLBnodes];
         D.f[dirNW  ] = &distributions[dirNW  *numberOfLBnodes];
         D.f[dirTE  ] = &distributions[dirTE  *numberOfLBnodes];
         D.f[dirBW  ] = &distributions[dirBW  *numberOfLBnodes];
         D.f[dirBE  ] = &distributions[dirBE  *numberOfLBnodes];
         D.f[dirTW  ] = &distributions[dirTW  *numberOfLBnodes];
         D.f[dirTN  ] = &distributions[dirTN  *numberOfLBnodes];
         D.f[dirBS  ] = &distributions[dirBS  *numberOfLBnodes];
         D.f[dirBN  ] = &distributions[dirBN  *numberOfLBnodes];
         D.f[dirTS  ] = &distributions[dirTS  *numberOfLBnodes];
         D.f[dirREST] = &distributions[dirREST*numberOfLBnodes];
         D.f[dirTNE ] = &distributions[dirTNE *numberOfLBnodes];
         D.f[dirTSW ] = &distributions[dirTSW *numberOfLBnodes];
         D.f[dirTSE ] = &distributions[dirTSE *numberOfLBnodes];
         D.f[dirTNW ] = &distributions[dirTNW *numberOfLBnodes];
         D.f[dirBNE ] = &distributions[dirBNE *numberOfLBnodes];
         D.f[dirBSW ] = &distributions[dirBSW *numberOfLBnodes];
         D.f[dirBSE ] = &distributions[dirBSE *numberOfLBnodes];
         D.f[dirBNW ] = &distributions[dirBNW *numberOfLBnodes];
      } 
      else
      {
         D.f[dirW   ] = &distributions[dirE   *numberOfLBnodes];
         D.f[dirE   ] = &distributions[dirW   *numberOfLBnodes];
         D.f[dirS   ] = &distributions[dirN   *numberOfLBnodes];
         D.f[dirN   ] = &distributions[dirS   *numberOfLBnodes];
         D.f[dirB   ] = &distributions[dirT   *numberOfLBnodes];
         D.f[dirT   ] = &distributions[dirB   *numberOfLBnodes];
         D.f[dirSW  ] = &distributions[dirNE  *numberOfLBnodes];
         D.f[dirNE  ] = &distributions[dirSW  *numberOfLBnodes];
         D.f[dirNW  ] = &distributions[dirSE  *numberOfLBnodes];
         D.f[dirSE  ] = &distributions[dirNW  *numberOfLBnodes];
         D.f[dirBW  ] = &distributions[dirTE  *numberOfLBnodes];
         D.f[dirTE  ] = &distributions[dirBW  *numberOfLBnodes];
         D.f[dirTW  ] = &distributions[dirBE  *numberOfLBnodes];
         D.f[dirBE  ] = &distributions[dirTW  *numberOfLBnodes];
         D.f[dirBS  ] = &distributions[dirTN  *numberOfLBnodes];
         D.f[dirTN  ] = &distributions[dirBS  *numberOfLBnodes];
         D.f[dirTS  ] = &distributions[dirBN  *numberOfLBnodes];
         D.f[dirBN  ] = &distributions[dirTS  *numberOfLBnodes];
         D.f[dirREST] = &distributions[dirREST*numberOfLBnodes];
         D.f[dirTNE ] = &distributions[dirBSW *numberOfLBnodes];
         D.f[dirTSW ] = &distributions[dirBNE *numberOfLBnodes];
         D.f[dirTSE ] = &distributions[dirBNW *numberOfLBnodes];
         D.f[dirTNW ] = &distributions[dirBSE *numberOfLBnodes];
         D.f[dirBNE ] = &distributions[dirTSW *numberOfLBnodes];
         D.f[dirBSW ] = &distributions[dirTNE *numberOfLBnodes];
         D.f[dirBSE ] = &distributions[dirTNW *numberOfLBnodes];
         D.f[dirBNW ] = &distributions[dirTSE *numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
         //(D.f[dirREST])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  
	  
	  //ToDo anders klammern !!!!!!
	  
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_E - f_W - c2o1 * drho * c2o27 * (c3o1*( vx1        ));
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W-m3+(f_E+f_W-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_E+f_W))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_W - f_E - c2o1 * drho * c2o27 * (c3o1*(-vx1        ));
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E-m3+(f_W+f_E-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_W+f_E))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_N - f_S - c2o1 * drho * c2o27 * (c3o1*( vx2        ));
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S-m3+(f_N+f_S-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_N+f_S))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_S - f_N - c2o1 * drho * c2o27 * (c3o1*(   -vx2     ));
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N-m3+(f_S+f_N-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_S+f_N))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_T - f_B - c2o1 * drho * c2o27 * (c3o1*(         vx3));
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B-m3+(f_T+f_B-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_T+f_B))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_B - f_T - c2o1 * drho * c2o27 * (c3o1*(        -vx3));
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T-m3+(f_B+f_T-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_B+f_T))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_NE - f_SW - c2o1 * drho * c1o54 * (c3o1*( vx1+vx2    ));
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW-m3+(f_NE+f_SW-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_NE+f_SW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_SW - f_NE - c2o1 * drho * c1o54 * (c3o1*(-vx1-vx2    ));
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE-m3+(f_SW+f_NE-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_SW+f_NE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_SE - f_NW - c2o1 * drho * c1o54 * (c3o1*( vx1-vx2    ));
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW-m3+(f_SE+f_NW-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_SE+f_NW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_NW - f_SE - c2o1 * drho * c1o54 * (c3o1*(-vx1+vx2    ));
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE-m3+(f_NW+f_SE-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_NW+f_SE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TE - f_BW - c2o1 * drho * c1o54 * (c3o1*( vx1    +vx3));
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW-m3+(f_TE+f_BW-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_TE+f_BW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BW - f_TE - c2o1 * drho * c1o54 * (c3o1*(-vx1    -vx3));
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE-m3+(f_BW+f_TE-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_BW+f_TE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BE - f_TW - c2o1 * drho * c1o54 * (c3o1*( vx1    -vx3));
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW-m3+(f_BE+f_TW-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_BE+f_TW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TW - f_BE - c2o1 * drho * c1o54 * (c3o1*(-vx1    +vx3));
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE-m3+(f_TW+f_BE-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_TW+f_BE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TN - f_BS - c2o1 * drho * c1o54 * (c3o1*(     vx2+vx3));
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS-m3+(f_TN+f_BS-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_TN+f_BS))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BS - f_TN - c2o1 * drho * c1o54 * (c3o1*(    -vx2-vx3));
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN-m3+(f_BS+f_TN-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_BS+f_TN))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BN - f_TS - c2o1 * drho * c1o54 * (c3o1*(     vx2-vx3));
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS-m3+(f_BN+f_TS-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_BN+f_TS))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TS - f_BN - c2o1 * drho * c1o54 * (c3o1*(    -vx2+vx3));
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN-m3+(f_TS+f_BN-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_TS+f_BN))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TNE - f_BSW - c2o1 * drho * c1o216 * (c3o1*( vx1+vx2+vx3));
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW-m3+(f_TNE+f_BSW-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BSW - f_TNE - c2o1 * drho * c1o216 * (c3o1*(-vx1-vx2-vx3));
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE-m3+(f_BSW+f_TNE-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BNE - f_TSW - c2o1 * drho * c1o216 * (c3o1*( vx1+vx2-vx3));
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW-m3+(f_BNE+f_TSW-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TSW - f_BNE - c2o1 * drho * c1o216 * (c3o1*(-vx1-vx2+vx3));
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE-m3+(f_TSW+f_BNE-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TSE - f_BNW - c2o1 * drho * c1o216 * (c3o1*( vx1-vx2+vx3));
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW-m3+(f_TSE+f_BNW-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BNW - f_TSE - c2o1 * drho * c1o216 * (c3o1*(-vx1+vx2-vx3));
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE-m3+(f_BNW+f_TSE-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BSE - f_TNW - c2o1 * drho * c1o216 * (c3o1*( vx1-vx2-vx3));
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW-m3+(f_BSE+f_TNW-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TNW - f_BSE - c2o1 * drho * c1o216 * (c3o1*(-vx1+vx2+vx3));
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE-m3+(f_TNW+f_BSE-c2o1*feq*omega)/(c1o1-omega))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDeviceIncompHighNu27(int inx,
												 int iny,
												 real* DD, 
												 int* k_Q, 
												 real* QQ,
												 unsigned int  numberOfBCnodes,
												 int numberOfNodes, 
												 real om1, 
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirREST] = &DD[dirREST*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirREST] = &DD[dirREST*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfNodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   * numberOfBCnodes];
      q_dirW   = &QQ[dirW   * numberOfBCnodes];
      q_dirN   = &QQ[dirN   * numberOfBCnodes];
      q_dirS   = &QQ[dirS   * numberOfBCnodes];
      q_dirT   = &QQ[dirT   * numberOfBCnodes];
      q_dirB   = &QQ[dirB   * numberOfBCnodes];
      q_dirNE  = &QQ[dirNE  * numberOfBCnodes];
      q_dirSW  = &QQ[dirSW  * numberOfBCnodes];
      q_dirSE  = &QQ[dirSE  * numberOfBCnodes];
      q_dirNW  = &QQ[dirNW  * numberOfBCnodes];
      q_dirTE  = &QQ[dirTE  * numberOfBCnodes];
      q_dirBW  = &QQ[dirBW  * numberOfBCnodes];
      q_dirBE  = &QQ[dirBE  * numberOfBCnodes];
      q_dirTW  = &QQ[dirTW  * numberOfBCnodes];
      q_dirTN  = &QQ[dirTN  * numberOfBCnodes];
      q_dirBS  = &QQ[dirBS  * numberOfBCnodes];
      q_dirBN  = &QQ[dirBN  * numberOfBCnodes];
      q_dirTS  = &QQ[dirTS  * numberOfBCnodes];
      q_dirTNE = &QQ[dirTNE * numberOfBCnodes];
      q_dirTSW = &QQ[dirTSW * numberOfBCnodes];
      q_dirTSE = &QQ[dirTSE * numberOfBCnodes];
      q_dirTNW = &QQ[dirTNW * numberOfBCnodes];
      q_dirBNE = &QQ[dirBNE * numberOfBCnodes];
      q_dirBSW = &QQ[dirBSW * numberOfBCnodes];
      q_dirBSE = &QQ[dirBSE * numberOfBCnodes];
      q_dirBNW = &QQ[dirBNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = k_Q[k];
      unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
            f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_E   = (D.f[dirE   ])[ke   ];
      f_W   = (D.f[dirW   ])[kw   ];
      f_N   = (D.f[dirN   ])[kn   ];
      f_S   = (D.f[dirS   ])[ks   ];
      f_T   = (D.f[dirT   ])[kt   ];
      f_B   = (D.f[dirB   ])[kb   ];
      f_NE  = (D.f[dirNE  ])[kne  ];
      f_SW  = (D.f[dirSW  ])[ksw  ];
      f_SE  = (D.f[dirSE  ])[kse  ];
      f_NW  = (D.f[dirNW  ])[knw  ];
      f_TE  = (D.f[dirTE  ])[kte  ];
      f_BW  = (D.f[dirBW  ])[kbw  ];
      f_BE  = (D.f[dirBE  ])[kbe  ];
      f_TW  = (D.f[dirTW  ])[ktw  ];
      f_TN  = (D.f[dirTN  ])[ktn  ];
      f_BS  = (D.f[dirBS  ])[kbs  ];
      f_BN  = (D.f[dirBN  ])[kbn  ];
      f_TS  = (D.f[dirTS  ])[kts  ];
      f_TNE = (D.f[dirTNE ])[ktne ];
      f_TSW = (D.f[dirTSW ])[ktsw ];
      f_TSE = (D.f[dirTSE ])[ktse ];
      f_TNW = (D.f[dirTNW ])[ktnw ];
      f_BNE = (D.f[dirBNE ])[kbne ];
      f_BSW = (D.f[dirBSW ])[kbsw ];
      f_BSE = (D.f[dirBSE ])[kbse ];
      f_BNW = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
				f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
				f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirREST])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W));// / (one + drho); 


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE))  + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                  (f_N - f_S));// / (one + drho); 

      vx3    =    (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                  (f_T - f_B));// / (one + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);// * (one + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirREST] = &DD[dirREST*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirREST] = &DD[dirREST*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
         //(D.f[dirREST])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX = c0o1;
      real VeloY = c0o1;
      real VeloZ = c0o1;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  
	  
	  //ToDo anders klammern !!!!!!
	  
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) /** (one + drho)*/-cu_sq); 
         (D.f[dirW])[kw]=((c1o1 - q) * f_E + q * ((f_E + f_W) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloX     )) / (q + c1o1) ;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) /** (one + drho)*/-cu_sq); 
         (D.f[dirE])[ke]=((c1o1 - q) * f_W + q * ((f_W + f_E) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloX     )) / (q + c1o1) ;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirS])[ks]=((c1o1 - q) * f_N + q * ((f_N + f_S) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloY     )) / (q + c1o1) ;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirN])[kn]=((c1o1 - q) * f_S + q * ((f_S + f_N) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloY     )) / (q + c1o1) ;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirB])[kb]=((c1o1 - q) * f_T + q * ((f_T + f_B) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloZ     )) / (q + c1o1) ;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirT])[kt]=((c1o1 - q) * f_B + q * ((f_B + f_T) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloZ     )) / (q + c1o1) ;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirSW])[ksw]=((c1o1 - q) * f_NE + q * ((f_NE + f_SW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX+VeloY)) / (q + c1o1) ;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirNE])[kne]=((c1o1 - q) * f_SW + q * ((f_SW + f_NE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX-VeloY)) / (q + c1o1) ;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirNW])[knw]=((c1o1 - q) * f_SE + q * ((f_SE + f_NW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX-VeloY)) / (q + c1o1) ;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirSE])[kse]=((c1o1 - q) * f_NW + q * ((f_NW + f_SE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX+VeloY)) / (q + c1o1) ;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBW])[kbw]=((c1o1 - q) * f_TE + q * ((f_TE + f_BW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTE])[kte]=((c1o1 - q) * f_BW + q * ((f_BW + f_TE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTW])[ktw]=((c1o1 - q) * f_BE + q * ((f_BE + f_TW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBE])[kbe]=((c1o1 - q) * f_TW + q * ((f_TW + f_BE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBS])[kbs]=((c1o1 - q) * f_TN + q * ((f_TN + f_BS) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTN])[ktn]=((c1o1 - q) * f_BS + q * ((f_BS + f_TN) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTS])[kts]=((c1o1 - q) * f_BN + q * ((f_BN + f_TS) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBN])[kbn]=((c1o1 - q) * f_TS + q * ((f_TS + f_BN) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBSW])[kbsw]=((c1o1 - q) * f_TNE + q * ((f_TNE + f_BSW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX+VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTNE])[ktne]=((c1o1 - q) * f_BSW + q * ((f_BSW + f_TNE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX-VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTSW])[ktsw]=((c1o1 - q) * f_BNE + q * ((f_BNE + f_TSW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX+VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBNE])[kbne]=((c1o1 - q) * f_TSW + q * ((f_TSW + f_BNE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX-VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBNW])[kbnw]=((c1o1 - q) * f_TSE + q * ((f_TSE + f_BNW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX-VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTSE])[ktse]=((c1o1 - q) * f_BNW + q * ((f_BNW + f_TSE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX+VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTNW])[ktnw]=((c1o1 - q) * f_BSE + q * ((f_BSE + f_TNW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX-VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBSE])[kbse]=((c1o1 - q) * f_TNW + q * ((f_TNW + f_BSE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX+VeloY+VeloZ)) / (q + c1o1) ;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDeviceCompHighNu27(
												 real* DD, 
												 int* k_Q, 
												 real* QQ,
												 unsigned int numberOfBCnodes, 
												 real om1, 
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirREST] = &DD[dirREST*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirREST] = &DD[dirREST*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   * numberOfBCnodes];
      q_dirW   = &QQ[dirW   * numberOfBCnodes];
      q_dirN   = &QQ[dirN   * numberOfBCnodes];
      q_dirS   = &QQ[dirS   * numberOfBCnodes];
      q_dirT   = &QQ[dirT   * numberOfBCnodes];
      q_dirB   = &QQ[dirB   * numberOfBCnodes];
      q_dirNE  = &QQ[dirNE  * numberOfBCnodes];
      q_dirSW  = &QQ[dirSW  * numberOfBCnodes];
      q_dirSE  = &QQ[dirSE  * numberOfBCnodes];
      q_dirNW  = &QQ[dirNW  * numberOfBCnodes];
      q_dirTE  = &QQ[dirTE  * numberOfBCnodes];
      q_dirBW  = &QQ[dirBW  * numberOfBCnodes];
      q_dirBE  = &QQ[dirBE  * numberOfBCnodes];
      q_dirTW  = &QQ[dirTW  * numberOfBCnodes];
      q_dirTN  = &QQ[dirTN  * numberOfBCnodes];
      q_dirBS  = &QQ[dirBS  * numberOfBCnodes];
      q_dirBN  = &QQ[dirBN  * numberOfBCnodes];
      q_dirTS  = &QQ[dirTS  * numberOfBCnodes];
      q_dirTNE = &QQ[dirTNE * numberOfBCnodes];
      q_dirTSW = &QQ[dirTSW * numberOfBCnodes];
      q_dirTSE = &QQ[dirTSE * numberOfBCnodes];
      q_dirTNW = &QQ[dirTNW * numberOfBCnodes];
      q_dirBNE = &QQ[dirBNE * numberOfBCnodes];
      q_dirBSW = &QQ[dirBSW * numberOfBCnodes];
      q_dirBSE = &QQ[dirBSE * numberOfBCnodes];
      q_dirBNW = &QQ[dirBNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = k_Q[k];
      unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
            f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_E   = (D.f[dirE   ])[ke   ];
      f_W   = (D.f[dirW   ])[kw   ];
      f_N   = (D.f[dirN   ])[kn   ];
      f_S   = (D.f[dirS   ])[ks   ];
      f_T   = (D.f[dirT   ])[kt   ];
      f_B   = (D.f[dirB   ])[kb   ];
      f_NE  = (D.f[dirNE  ])[kne  ];
      f_SW  = (D.f[dirSW  ])[ksw  ];
      f_SE  = (D.f[dirSE  ])[kse  ];
      f_NW  = (D.f[dirNW  ])[knw  ];
      f_TE  = (D.f[dirTE  ])[kte  ];
      f_BW  = (D.f[dirBW  ])[kbw  ];
      f_BE  = (D.f[dirBE  ])[kbe  ];
      f_TW  = (D.f[dirTW  ])[ktw  ];
      f_TN  = (D.f[dirTN  ])[ktn  ];
      f_BS  = (D.f[dirBS  ])[kbs  ];
      f_BN  = (D.f[dirBN  ])[kbn  ];
      f_TS  = (D.f[dirTS  ])[kts  ];
      f_TNE = (D.f[dirTNE ])[ktne ];
      f_TSW = (D.f[dirTSW ])[ktsw ];
      f_TSE = (D.f[dirTSE ])[ktse ];
      f_TNW = (D.f[dirTNW ])[ktnw ];
      f_BNE = (D.f[dirBNE ])[kbne ];
      f_BSW = (D.f[dirBSW ])[kbsw ];
      f_BSE = (D.f[dirBSE ])[kbse ];
      f_BNW = (D.f[dirBNW ])[kbnw ];
      //f_W    = (D.f[dirE   ])[ke   ];
      //f_E    = (D.f[dirW   ])[kw   ];
      //f_S    = (D.f[dirN   ])[kn   ];
      //f_N    = (D.f[dirS   ])[ks   ];
      //f_B    = (D.f[dirT   ])[kt   ];
      //f_T    = (D.f[dirB   ])[kb   ];
      //f_SW   = (D.f[dirNE  ])[kne  ];
      //f_NE   = (D.f[dirSW  ])[ksw  ];
      //f_NW   = (D.f[dirSE  ])[kse  ];
      //f_SE   = (D.f[dirNW  ])[knw  ];
      //f_BW   = (D.f[dirTE  ])[kte  ];
      //f_TE   = (D.f[dirBW  ])[kbw  ];
      //f_TW   = (D.f[dirBE  ])[kbe  ];
      //f_BE   = (D.f[dirTW  ])[ktw  ];
      //f_BS   = (D.f[dirTN  ])[ktn  ];
      //f_TN   = (D.f[dirBS  ])[kbs  ];
      //f_TS   = (D.f[dirBN  ])[kbn  ];
      //f_BN   = (D.f[dirTS  ])[kts  ];
      //f_BSW  = (D.f[dirTNE ])[ktne ];
      //f_BNE  = (D.f[dirTSW ])[ktsw ];
      //f_BNW  = (D.f[dirTSE ])[ktse ];
      //f_BSE  = (D.f[dirTNW ])[ktnw ];
      //f_TSW  = (D.f[dirBNE ])[kbne ];
      //f_TNE  = (D.f[dirBSW ])[kbsw ];
      //f_TNW  = (D.f[dirBSE ])[kbse ];
      //f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
				f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
				f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirREST])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE))  + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                  (f_N - f_S)) / (c1o1 + drho); 

      vx3    =    (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                  (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirREST] = &DD[dirREST*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirREST] = &DD[dirREST*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
         //(D.f[dirREST])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX = c0o1;
      real VeloY = c0o1;
      real VeloZ = c0o1;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  
	  
	  //ToDo anders klammern !!!!!!
	  
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=((c1o1 - q) * f_E + q * ((f_E + f_W) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloX     )) / (q + c1o1) ;
         //(D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-f_W+(f_E+f_W-two*feq*om1)/(one-om1))*c1o2+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q) - c2over27 * drho;
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=((c1o1 - q) * f_W + q * ((f_W + f_E) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloX     )) / (q + c1o1) ;
         //(D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-f_E+(f_W+f_E-two*feq*om1)/(one-om1))*c1o2+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q) - c2over27 * drho;
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=((c1o1 - q) * f_N + q * ((f_N + f_S) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloY     )) / (q + c1o1) ;
         //(D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-f_S+(f_N+f_S-two*feq*om1)/(one-om1))*c1o2+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q) - c2over27 * drho;
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=((c1o1 - q) * f_S + q * ((f_S + f_N) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloY     )) / (q + c1o1) ;
         //(D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-f_N+(f_S+f_N-two*feq*om1)/(one-om1))*c1o2+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q) - c2over27 * drho;
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=((c1o1 - q) * f_T + q * ((f_T + f_B) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloZ     )) / (q + c1o1) ;
         //(D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-f_B+(f_T+f_B-two*feq*om1)/(one-om1))*c1o2+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q) - c2over27 * drho;
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=((c1o1 - q) * f_B + q * ((f_B + f_T) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloZ     )) / (q + c1o1) ;
         //(D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-f_T+(f_B+f_T-two*feq*om1)/(one-om1))*c1o2+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q) - c2over27 * drho;
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=((c1o1 - q) * f_NE + q * ((f_NE + f_SW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX+VeloY)) / (q + c1o1) ;
         //(D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-f_SW+(f_NE+f_SW-two*feq*om1)/(one-om1))*c1o2+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q) - c1over54 * drho;
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=((c1o1 - q) * f_SW + q * ((f_SW + f_NE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX-VeloY)) / (q + c1o1) ;
         //(D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-f_NE+(f_SW+f_NE-two*feq*om1)/(one-om1))*c1o2+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q) - c1over54 * drho;
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=((c1o1 - q) * f_SE + q * ((f_SE + f_NW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX-VeloY)) / (q + c1o1) ;
         //(D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-f_NW+(f_SE+f_NW-two*feq*om1)/(one-om1))*c1o2+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q) - c1over54 * drho;
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=((c1o1 - q) * f_NW + q * ((f_NW + f_SE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX+VeloY)) / (q + c1o1) ;
         //(D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-f_SE+(f_NW+f_SE-two*feq*om1)/(one-om1))*c1o2+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q) - c1over54 * drho;
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=((c1o1 - q) * f_TE + q * ((f_TE + f_BW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-f_BW+(f_TE+f_BW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=((c1o1 - q) * f_BW + q * ((f_BW + f_TE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-f_TE+(f_BW+f_TE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=((c1o1 - q) * f_BE + q * ((f_BE + f_TW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-f_TW+(f_BE+f_TW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=((c1o1 - q) * f_TW + q * ((f_TW + f_BE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-f_BE+(f_TW+f_BE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=((c1o1 - q) * f_TN + q * ((f_TN + f_BS) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-f_BS+(f_TN+f_BS-two*feq*om1)/(one-om1))*c1o2+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=((c1o1 - q) * f_BS + q * ((f_BS + f_TN) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-f_TN+(f_BS+f_TN-two*feq*om1)/(one-om1))*c1o2+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=((c1o1 - q) * f_BN + q * ((f_BN + f_TS) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-f_TS+(f_BN+f_TS-two*feq*om1)/(one-om1))*c1o2+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=((c1o1 - q) * f_TS + q * ((f_TS + f_BN) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-f_BN+(f_TS+f_BN-two*feq*om1)/(one-om1))*c1o2+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=((c1o1 - q) * f_TNE + q * ((f_TNE + f_BSW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX+VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=((c1o1 - q) * f_BSW + q * ((f_BSW + f_TNE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX-VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=((c1o1 - q) * f_BNE + q * ((f_BNE + f_TSW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX+VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=((c1o1 - q) * f_TSW + q * ((f_TSW + f_BNE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX-VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=((c1o1 - q) * f_TSE + q * ((f_TSE + f_BNW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX-VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=((c1o1 - q) * f_BNW + q * ((f_BNW + f_TSE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX+VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=((c1o1 - q) * f_BSE + q * ((f_BSE + f_TNW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX-VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=((c1o1 - q) * f_TNW + q * ((f_TNW + f_BSE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX+VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDeviceComp27(
										 real* distributions, 
										 int* subgridDistanceIndices, 
										 real* subgridDistances,
										 unsigned int numberOfBCnodes, 
										 real omega, 
										 unsigned int* neighborX,
										 unsigned int* neighborY,
										 unsigned int* neighborZ,
										 unsigned int numberOfLBnodes, 
										 bool isEvenTimestep)
{
   //////////////////////////////////////////////////////////////////////////
   //! The velocity boundary condition is executed in the following steps
   //!
   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned  x = threadIdx.x;  // global x-index 
   const unsigned  y = blockIdx.x;   // global y-index 
   const unsigned  z = blockIdx.y;   // global z-index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;

   if(k < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[k];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[dirE   ])[ke   ];
      real f_E    = (dist.f[dirW   ])[kw   ];
      real f_S    = (dist.f[dirN   ])[kn   ];
      real f_N    = (dist.f[dirS   ])[ks   ];
      real f_B    = (dist.f[dirT   ])[kt   ];
      real f_T    = (dist.f[dirB   ])[kb   ];
      real f_SW   = (dist.f[dirNE  ])[kne  ];
      real f_NE   = (dist.f[dirSW  ])[ksw  ];
      real f_NW   = (dist.f[dirSE  ])[kse  ];
      real f_SE   = (dist.f[dirNW  ])[knw  ];
      real f_BW   = (dist.f[dirTE  ])[kte  ];
      real f_TE   = (dist.f[dirBW  ])[kbw  ];
      real f_TW   = (dist.f[dirBE  ])[kbe  ];
      real f_BE   = (dist.f[dirTW  ])[ktw  ];
      real f_BS   = (dist.f[dirTN  ])[ktn  ];
      real f_TN   = (dist.f[dirBS  ])[kbs  ];
      real f_TS   = (dist.f[dirBN  ])[kbn  ];
      real f_BN   = (dist.f[dirTS  ])[kts  ];
      real f_BSW  = (dist.f[dirTNE ])[ktne ];
      real f_BNE  = (dist.f[dirTSW ])[ktsw ];
      real f_BNW  = (dist.f[dirTSE ])[ktse ];
      real f_BSE  = (dist.f[dirTNW ])[ktnw ];
      real f_TSW  = (dist.f[dirBNE ])[kbne ];
      real f_TNE  = (dist.f[dirBSW ])[kbsw ];
      real f_TNW  = (dist.f[dirBSE ])[kbse ];
      real f_TSE  = (dist.f[dirBNW ])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[dirREST])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);          

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho); 

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

       ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      real feq, q, velocityLB;
      q = (subgridD.q[dirE])[k];
      if (q>=c0o1 && q<=c1o1) // only update distribution for q between zero and one
      {
         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirW])[kw] = getInterpolatedDistributionForNoSlipBC(q, f_E, f_W, feq, omega);
      }

      q = (subgridD.q[dirW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirE])[ke] = getInterpolatedDistributionForNoSlipBC(q, f_W, f_E, feq, omega);
      }

      q = (subgridD.q[dirN])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirS])[ks] = getInterpolatedDistributionForNoSlipBC(q, f_N, f_S, feq, omega);
      }

      q = (subgridD.q[dirS])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirN])[kn] = getInterpolatedDistributionForNoSlipBC(q, f_S, f_N, feq, omega);
      }

      q = (subgridD.q[dirT])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirB])[kb] = getInterpolatedDistributionForNoSlipBC(q, f_T, f_B, feq, omega);
      }

      q = (subgridD.q[dirB])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirT])[kt] = getInterpolatedDistributionForNoSlipBC(q, f_B, f_T, feq, omega);
      }

      q = (subgridD.q[dirNE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirSW])[ksw] = getInterpolatedDistributionForNoSlipBC(q, f_NE, f_SW, feq, omega);
      }

      q = (subgridD.q[dirSW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirNE])[kne] = getInterpolatedDistributionForNoSlipBC(q, f_SW, f_NE, feq, omega);
      }

      q = (subgridD.q[dirSE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirNW])[knw] = getInterpolatedDistributionForNoSlipBC(q, f_SE, f_NW, feq, omega);
      }

      q = (subgridD.q[dirNW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirSE])[kse] = getInterpolatedDistributionForNoSlipBC(q, f_NW, f_SE, feq, omega);
      }

      q = (subgridD.q[dirTE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirBW])[kbw] = getInterpolatedDistributionForNoSlipBC(q, f_TE, f_BW, feq, omega);
      }

      q = (subgridD.q[dirBW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirTE])[kte] = getInterpolatedDistributionForNoSlipBC(q, f_BW, f_TE, feq, omega);
      }

      q = (subgridD.q[dirBE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirTW])[ktw] = getInterpolatedDistributionForNoSlipBC(q, f_BE, f_TW, feq, omega);
      }

      q = (subgridD.q[dirTW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirBE])[kbe] = getInterpolatedDistributionForNoSlipBC(q, f_TW, f_BE, feq, omega);
      }

      q = (subgridD.q[dirTN])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirBS])[kbs] = getInterpolatedDistributionForNoSlipBC(q, f_TN, f_BS, feq, omega);
      }

      q = (subgridD.q[dirBS])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirTN])[ktn] = getInterpolatedDistributionForNoSlipBC(q, f_BS, f_TN, feq, omega);
      }

      q = (subgridD.q[dirBN])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirTS])[kts] = getInterpolatedDistributionForNoSlipBC(q, f_BN, f_TS, feq, omega);
      }

      q = (subgridD.q[dirTS])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirBN])[kbn] = getInterpolatedDistributionForNoSlipBC(q, f_TS, f_BN, feq, omega);
      }

      q = (subgridD.q[dirTNE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirBSW])[kbsw] = getInterpolatedDistributionForNoSlipBC(q, f_TNE, f_BSW, feq, omega);
      }

      q = (subgridD.q[dirBSW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirTNE])[ktne] = getInterpolatedDistributionForNoSlipBC(q, f_BSW, f_TNE, feq, omega);
      }

      q = (subgridD.q[dirBNE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirTSW])[ktsw] = getInterpolatedDistributionForNoSlipBC(q, f_BNE, f_TSW, feq, omega);
      }

      q = (subgridD.q[dirTSW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirBNE])[kbne] = getInterpolatedDistributionForNoSlipBC(q, f_TSW, f_BNE, feq, omega);
      }

      q = (subgridD.q[dirTSE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirBNW])[kbnw] = getInterpolatedDistributionForNoSlipBC(q, f_TSE, f_BNW, feq, omega);
      }

      q = (subgridD.q[dirBNW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirTSE])[ktse] = getInterpolatedDistributionForNoSlipBC(q, f_BNW, f_TSE, feq, omega);
      }

      q = (subgridD.q[dirBSE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirTNW])[ktnw] = getInterpolatedDistributionForNoSlipBC(q, f_BSE, f_TNW, feq, omega);
      }

      q = (subgridD.q[dirTNW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirBSE])[kbse] = getInterpolatedDistributionForNoSlipBC(q, f_TNW, f_BSE, feq, omega);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDevice27(real* distributions, 
                                     int* subgridDistanceIndices, 
                                     real* subgridDistances,
                                     unsigned int numberOfBCnodes, 
                                     real omega, 
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int numberOfLBnodes, 
                                     bool isEvenTimestep)
{
   //////////////////////////////////////////////////////////////////////////
   //! The no-slip boundary condition is executed in the following steps
   //!
   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned  x = threadIdx.x;  // global x-index 
   const unsigned  y = blockIdx.x;   // global y-index 
   const unsigned  z = blockIdx.y;   // global z-index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;

   //////////////////////////////////////////////////////////////////////////
   //! - Run for all indices in size of boundary condition (numberOfBCnodes)
   //!
   if(k < numberOfBCnodes)
   {

      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[k];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[dirE   ])[ke   ];
      real f_E    = (dist.f[dirW   ])[kw   ];
      real f_S    = (dist.f[dirN   ])[kn   ];
      real f_N    = (dist.f[dirS   ])[ks   ];
      real f_B    = (dist.f[dirT   ])[kt   ];
      real f_T    = (dist.f[dirB   ])[kb   ];
      real f_SW   = (dist.f[dirNE  ])[kne  ];
      real f_NE   = (dist.f[dirSW  ])[ksw  ];
      real f_NW   = (dist.f[dirSE  ])[kse  ];
      real f_SE   = (dist.f[dirNW  ])[knw  ];
      real f_BW   = (dist.f[dirTE  ])[kte  ];
      real f_TE   = (dist.f[dirBW  ])[kbw  ];
      real f_TW   = (dist.f[dirBE  ])[kbe  ];
      real f_BE   = (dist.f[dirTW  ])[ktw  ];
      real f_BS   = (dist.f[dirTN  ])[ktn  ];
      real f_TN   = (dist.f[dirBS  ])[kbs  ];
      real f_TS   = (dist.f[dirBN  ])[kbn  ];
      real f_BN   = (dist.f[dirTS  ])[kts  ];
      real f_BSW  = (dist.f[dirTNE ])[ktne ];
      real f_BNE  = (dist.f[dirTSW ])[ktsw ];
      real f_BNW  = (dist.f[dirTSE ])[ktse ];
      real f_BSE  = (dist.f[dirTNW ])[ktnw ];
      real f_TSW  = (dist.f[dirBNE ])[kbne ];
      real f_TNE  = (dist.f[dirBSW ])[kbsw ];
      real f_TNW  = (dist.f[dirBSE ])[kbse ];
      real f_TSE  = (dist.f[dirBNW ])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[dirREST])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W));          

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)); 

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)); 

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real feq, q, velocityLB;
      q = (subgridD.q[dirE])[k];
      if (q>=c0o1 && q<=c1o1) // only update distribution for q between zero and one
      {
         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirW])[kw] = getInterpolatedDistributionForNoSlipBC(q, f_E, f_W, feq, omega);
      }

      q = (subgridD.q[dirW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirE])[ke] = getInterpolatedDistributionForNoSlipBC(q, f_W, f_E, feq, omega);
      }

      q = (subgridD.q[dirN])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirS])[ks] = getInterpolatedDistributionForNoSlipBC(q, f_N, f_S, feq, omega);
      }

      q = (subgridD.q[dirS])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirN])[kn] = getInterpolatedDistributionForNoSlipBC(q, f_S, f_N, feq, omega);
      }

      q = (subgridD.q[dirT])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirB])[kb] = getInterpolatedDistributionForNoSlipBC(q, f_T, f_B, feq, omega);
      }

      q = (subgridD.q[dirB])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         (dist.f[dirT])[kt] = getInterpolatedDistributionForNoSlipBC(q, f_B, f_T, feq, omega);
      }

      q = (subgridD.q[dirNE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirSW])[ksw] = getInterpolatedDistributionForNoSlipBC(q, f_NE, f_SW, feq, omega);
      }

      q = (subgridD.q[dirSW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirNE])[kne] = getInterpolatedDistributionForNoSlipBC(q, f_SW, f_NE, feq, omega);
      }

      q = (subgridD.q[dirSE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirNW])[knw] = getInterpolatedDistributionForNoSlipBC(q, f_SE, f_NW, feq, omega);
      }

      q = (subgridD.q[dirNW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirSE])[kse] = getInterpolatedDistributionForNoSlipBC(q, f_NW, f_SE, feq, omega);
      }

      q = (subgridD.q[dirTE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirBW])[kbw] = getInterpolatedDistributionForNoSlipBC(q, f_TE, f_BW, feq, omega);
      }

      q = (subgridD.q[dirBW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirTE])[kte] = getInterpolatedDistributionForNoSlipBC(q, f_BW, f_TE, feq, omega);
      }

      q = (subgridD.q[dirBE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirTW])[ktw] = getInterpolatedDistributionForNoSlipBC(q, f_BE, f_TW, feq, omega);
      }

      q = (subgridD.q[dirTW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirBE])[kbe] = getInterpolatedDistributionForNoSlipBC(q, f_TW, f_BE, feq, omega);
      }

      q = (subgridD.q[dirTN])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirBS])[kbs] = getInterpolatedDistributionForNoSlipBC(q, f_TN, f_BS, feq, omega);
      }

      q = (subgridD.q[dirBS])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirTN])[ktn] = getInterpolatedDistributionForNoSlipBC(q, f_BS, f_TN, feq, omega);
      }

      q = (subgridD.q[dirBN])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirTS])[kts] = getInterpolatedDistributionForNoSlipBC(q, f_BN, f_TS, feq, omega);
      }

      q = (subgridD.q[dirTS])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         (dist.f[dirBN])[kbn] = getInterpolatedDistributionForNoSlipBC(q, f_TS, f_BN, feq, omega);
      }

      q = (subgridD.q[dirTNE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirBSW])[kbsw] = getInterpolatedDistributionForNoSlipBC(q, f_TNE, f_BSW, feq, omega);
      }

      q = (subgridD.q[dirBSW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirTNE])[ktne] = getInterpolatedDistributionForNoSlipBC(q, f_BSW, f_TNE, feq, omega);
      }

      q = (subgridD.q[dirBNE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirTSW])[ktsw] = getInterpolatedDistributionForNoSlipBC(q, f_BNE, f_TSW, feq, omega);
      }

      q = (subgridD.q[dirTSW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirBNE])[kbne] = getInterpolatedDistributionForNoSlipBC(q, f_TSW, f_BNE, feq, omega);
      }

      q = (subgridD.q[dirTSE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirBNW])[kbnw] = getInterpolatedDistributionForNoSlipBC(q, f_TSE, f_BNW, feq, omega);
      }

      q = (subgridD.q[dirBNW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirTSE])[ktse] = getInterpolatedDistributionForNoSlipBC(q, f_BNW, f_TSE, feq, omega);
      }

      q = (subgridD.q[dirBSE])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirTNW])[ktnw] = getInterpolatedDistributionForNoSlipBC(q, f_BSE, f_TNW, feq, omega);
      }

      q = (subgridD.q[dirTNW])[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         (dist.f[dirBSE])[kbse] = getInterpolatedDistributionForNoSlipBC(q, f_TNW, f_BSE, feq, omega);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void BBDevice27(real* distributions, 
                                     int* subgridDistanceIndices, 
                                     real* subgridDistances,
                                     unsigned int numberOfBCnodes, 
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int numberOfLBnodes, 
                                     bool isEvenTimestep)
{
   //////////////////////////////////////////////////////////////////////////
   //! The no-slip boundary condition is executed in the following steps
   //!
   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned  x = threadIdx.x;   // global x-index
   const unsigned  y = blockIdx.x;    // global y-index
   const unsigned  z = blockIdx.y;    // global z-index

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;

   //////////////////////////////////////////////////////////////////////////
   // run for all indices in size of boundary condition (numberOfBCnodes)
   if(k < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[k];
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[dirE   ])[ke   ];
      real f_E    = (dist.f[dirW   ])[kw   ];
      real f_S    = (dist.f[dirN   ])[kn   ];
      real f_N    = (dist.f[dirS   ])[ks   ];
      real f_B    = (dist.f[dirT   ])[kt   ];
      real f_T    = (dist.f[dirB   ])[kb   ];
      real f_SW   = (dist.f[dirNE  ])[kne  ];
      real f_NE   = (dist.f[dirSW  ])[ksw  ];
      real f_NW   = (dist.f[dirSE  ])[kse  ];
      real f_SE   = (dist.f[dirNW  ])[knw  ];
      real f_BW   = (dist.f[dirTE  ])[kte  ];
      real f_TE   = (dist.f[dirBW  ])[kbw  ];
      real f_TW   = (dist.f[dirBE  ])[kbe  ];
      real f_BE   = (dist.f[dirTW  ])[ktw  ];
      real f_BS   = (dist.f[dirTN  ])[ktn  ];
      real f_TN   = (dist.f[dirBS  ])[kbs  ];
      real f_TS   = (dist.f[dirBN  ])[kbn  ];
      real f_BN   = (dist.f[dirTS  ])[kts  ];
      real f_BSW  = (dist.f[dirTNE ])[ktne ];
      real f_BNE  = (dist.f[dirTSW ])[ktsw ];
      real f_BNW  = (dist.f[dirTSE ])[ktse ];
      real f_BSE  = (dist.f[dirTNW ])[ktnw ];
      real f_TSW  = (dist.f[dirBNE ])[kbne ];
      real f_TNE  = (dist.f[dirBSW ])[kbsw ];
      real f_TNW  = (dist.f[dirBSE ])[kbse ];
      real f_TSE  = (dist.f[dirBNW ])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - rewrite distributions if there is a sub-grid distance (q) in same direction
      real q;
      q = (subgridD.q[dirE  ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirW  ])[kw  ]=f_E  ;
      q = (subgridD.q[dirW  ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirE  ])[ke  ]=f_W  ;
      q = (subgridD.q[dirN  ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirS  ])[ks  ]=f_N  ;
      q = (subgridD.q[dirS  ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirN  ])[kn  ]=f_S  ;
      q = (subgridD.q[dirT  ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirB  ])[kb  ]=f_T  ;
      q = (subgridD.q[dirB  ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirT  ])[kt  ]=f_B  ;
      q = (subgridD.q[dirNE ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirSW ])[ksw ]=f_NE ;
      q = (subgridD.q[dirSW ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirNE ])[kne ]=f_SW ;
      q = (subgridD.q[dirSE ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirNW ])[knw ]=f_SE ;
      q = (subgridD.q[dirNW ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirSE ])[kse ]=f_NW ;
      q = (subgridD.q[dirTE ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirBW ])[kbw ]=f_TE ;
      q = (subgridD.q[dirBW ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirTE ])[kte ]=f_BW ;
      q = (subgridD.q[dirBE ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirTW ])[ktw ]=f_BE ;
      q = (subgridD.q[dirTW ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirBE ])[kbe ]=f_TW ;
      q = (subgridD.q[dirTN ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirBS ])[kbs ]=f_TN ;
      q = (subgridD.q[dirBS ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirTN ])[ktn ]=f_BS ;
      q = (subgridD.q[dirBN ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirTS ])[kts ]=f_BN ;
      q = (subgridD.q[dirTS ])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirBN ])[kbn ]=f_TS ;
      q = (subgridD.q[dirTNE])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirBSW])[kbsw]=f_TNE;
      q = (subgridD.q[dirBSW])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirTNE])[ktne]=f_BSW;
      q = (subgridD.q[dirBNE])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirTSW])[ktsw]=f_BNE;
      q = (subgridD.q[dirTSW])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirBNE])[kbne]=f_TSW;
      q = (subgridD.q[dirTSE])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirBNW])[kbnw]=f_TSE;
      q = (subgridD.q[dirBNW])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirTSE])[ktse]=f_BNW;
      q = (subgridD.q[dirBSE])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirTNW])[ktnw]=f_BSE;
      q = (subgridD.q[dirTNW])[k];   if (q>=c0o1 && q<=c1o1)    (dist.f[dirBSE])[kbse]=f_TNW;
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

