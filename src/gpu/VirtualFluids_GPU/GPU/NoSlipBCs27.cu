#include "hip/hip_runtime.h"
//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ / 
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;

//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDevice3rdMomentsComp27(  int inx,
													 int iny,
													 real* DD, 
													 int* k_Q, 
													 real* QQ,
													 unsigned int numberOfBCnodes, 
													 real om1, 
													 unsigned int* neighborX,
													 unsigned int* neighborY,
													 unsigned int* neighborZ,
													 unsigned int size_Mat, 
													 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k < numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   * numberOfBCnodes];
      q_dirW   = &QQ[dirW   * numberOfBCnodes];
      q_dirN   = &QQ[dirN   * numberOfBCnodes];
      q_dirS   = &QQ[dirS   * numberOfBCnodes];
      q_dirT   = &QQ[dirT   * numberOfBCnodes];
      q_dirB   = &QQ[dirB   * numberOfBCnodes];
      q_dirNE  = &QQ[dirNE  * numberOfBCnodes];
      q_dirSW  = &QQ[dirSW  * numberOfBCnodes];
      q_dirSE  = &QQ[dirSE  * numberOfBCnodes];
      q_dirNW  = &QQ[dirNW  * numberOfBCnodes];
      q_dirTE  = &QQ[dirTE  * numberOfBCnodes];
      q_dirBW  = &QQ[dirBW  * numberOfBCnodes];
      q_dirBE  = &QQ[dirBE  * numberOfBCnodes];
      q_dirTW  = &QQ[dirTW  * numberOfBCnodes];
      q_dirTN  = &QQ[dirTN  * numberOfBCnodes];
      q_dirBS  = &QQ[dirBS  * numberOfBCnodes];
      q_dirBN  = &QQ[dirBN  * numberOfBCnodes];
      q_dirTS  = &QQ[dirTS  * numberOfBCnodes];
      q_dirTNE = &QQ[dirTNE * numberOfBCnodes];
      q_dirTSW = &QQ[dirTSW * numberOfBCnodes];
      q_dirTSE = &QQ[dirTSE * numberOfBCnodes];
      q_dirTNW = &QQ[dirTNW * numberOfBCnodes];
      q_dirBNE = &QQ[dirBNE * numberOfBCnodes];
      q_dirBSW = &QQ[dirBSW * numberOfBCnodes];
      q_dirBSE = &QQ[dirBSE * numberOfBCnodes];
      q_dirBNW = &QQ[dirBNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = k_Q[k];
      unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
            f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dirE   ])[ke   ];
      f_E    = (D.f[dirW   ])[kw   ];
      f_S    = (D.f[dirN   ])[kn   ];
      f_N    = (D.f[dirS   ])[ks   ];
      f_B    = (D.f[dirT   ])[kt   ];
      f_T    = (D.f[dirB   ])[kb   ];
      f_SW   = (D.f[dirNE  ])[kne  ];
      f_NE   = (D.f[dirSW  ])[ksw  ];
      f_NW   = (D.f[dirSE  ])[kse  ];
      f_SE   = (D.f[dirNW  ])[knw  ];
      f_BW   = (D.f[dirTE  ])[kte  ];
      f_TE   = (D.f[dirBW  ])[kbw  ];
      f_TW   = (D.f[dirBE  ])[kbe  ];
      f_BE   = (D.f[dirTW  ])[ktw  ];
      f_BS   = (D.f[dirTN  ])[ktn  ];
      f_TN   = (D.f[dirBS  ])[kbs  ];
      f_TS   = (D.f[dirBN  ])[kbn  ];
      f_BN   = (D.f[dirTS  ])[kts  ];
      f_BSW  = (D.f[dirTNE ])[ktne ];
      f_BNE  = (D.f[dirTSW ])[ktsw ];
      f_BNW  = (D.f[dirTSE ])[ktse ];
      f_BSE  = (D.f[dirTNW ])[ktnw ];
      f_TSW  = (D.f[dirBNE ])[kbne ];
      f_TNE  = (D.f[dirBSW ])[kbsw ];
      f_TNW  = (D.f[dirBSE ])[kbse ];
      f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q, m3;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
				f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
				f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE))  + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                  (f_N - f_S)) / (c1o1 + drho); 

      vx3    =    (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                  (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
         //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  
	  
	  //ToDo anders klammern !!!!!!
	  
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_E - f_W - c2o1 * drho * c2o27 * (c3o1*( vx1        ));
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W-m3+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_W - f_E - c2o1 * drho * c2o27 * (c3o1*(-vx1        ));
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E-m3+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_N - f_S - c2o1 * drho * c2o27 * (c3o1*( vx2        ));
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S-m3+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_S - f_N - c2o1 * drho * c2o27 * (c3o1*(   -vx2     ));
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N-m3+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_T - f_B - c2o1 * drho * c2o27 * (c3o1*(         vx3));
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B-m3+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_B - f_T - c2o1 * drho * c2o27 * (c3o1*(        -vx3));
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T-m3+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_NE - f_SW - c2o1 * drho * c1o54 * (c3o1*( vx1+vx2    ));
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW-m3+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_SW - f_NE - c2o1 * drho * c1o54 * (c3o1*(-vx1-vx2    ));
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE-m3+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_SE - f_NW - c2o1 * drho * c1o54 * (c3o1*( vx1-vx2    ));
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW-m3+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_NW - f_SE - c2o1 * drho * c1o54 * (c3o1*(-vx1+vx2    ));
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE-m3+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TE - f_BW - c2o1 * drho * c1o54 * (c3o1*( vx1    +vx3));
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW-m3+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BW - f_TE - c2o1 * drho * c1o54 * (c3o1*(-vx1    -vx3));
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE-m3+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BE - f_TW - c2o1 * drho * c1o54 * (c3o1*( vx1    -vx3));
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW-m3+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TW - f_BE - c2o1 * drho * c1o54 * (c3o1*(-vx1    +vx3));
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE-m3+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TN - f_BS - c2o1 * drho * c1o54 * (c3o1*(     vx2+vx3));
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS-m3+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BS - f_TN - c2o1 * drho * c1o54 * (c3o1*(    -vx2-vx3));
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN-m3+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BN - f_TS - c2o1 * drho * c1o54 * (c3o1*(     vx2-vx3));
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS-m3+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TS - f_BN - c2o1 * drho * c1o54 * (c3o1*(    -vx2+vx3));
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN-m3+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TNE - f_BSW - c2o1 * drho * c1o216 * (c3o1*( vx1+vx2+vx3));
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW-m3+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BSW - f_TNE - c2o1 * drho * c1o216 * (c3o1*(-vx1-vx2-vx3));
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE-m3+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BNE - f_TSW - c2o1 * drho * c1o216 * (c3o1*( vx1+vx2-vx3));
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW-m3+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TSW - f_BNE - c2o1 * drho * c1o216 * (c3o1*(-vx1-vx2+vx3));
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE-m3+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TSE - f_BNW - c2o1 * drho * c1o216 * (c3o1*( vx1-vx2+vx3));
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW-m3+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BNW - f_TSE - c2o1 * drho * c1o216 * (c3o1*(-vx1+vx2-vx3));
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE-m3+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_BSE - f_TNW - c2o1 * drho * c1o216 * (c3o1*( vx1-vx2-vx3));
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW-m3+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
		 m3 = f_TNW - f_BSE - c2o1 * drho * c1o216 * (c3o1*(-vx1+vx2+vx3));
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE-m3+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q)+(m3*c1o2);
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDeviceIncompHighNu27(int inx,
												 int iny,
												 real* DD, 
												 int* k_Q, 
												 real* QQ,
												 unsigned int sizeQ,
												 int numberOfNodes, 
												 real om1, 
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfNodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = k_Q[k];
      unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
            f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_E   = (D.f[dirE   ])[ke   ];
      f_W   = (D.f[dirW   ])[kw   ];
      f_N   = (D.f[dirN   ])[kn   ];
      f_S   = (D.f[dirS   ])[ks   ];
      f_T   = (D.f[dirT   ])[kt   ];
      f_B   = (D.f[dirB   ])[kb   ];
      f_NE  = (D.f[dirNE  ])[kne  ];
      f_SW  = (D.f[dirSW  ])[ksw  ];
      f_SE  = (D.f[dirSE  ])[kse  ];
      f_NW  = (D.f[dirNW  ])[knw  ];
      f_TE  = (D.f[dirTE  ])[kte  ];
      f_BW  = (D.f[dirBW  ])[kbw  ];
      f_BE  = (D.f[dirBE  ])[kbe  ];
      f_TW  = (D.f[dirTW  ])[ktw  ];
      f_TN  = (D.f[dirTN  ])[ktn  ];
      f_BS  = (D.f[dirBS  ])[kbs  ];
      f_BN  = (D.f[dirBN  ])[kbn  ];
      f_TS  = (D.f[dirTS  ])[kts  ];
      f_TNE = (D.f[dirTNE ])[ktne ];
      f_TSW = (D.f[dirTSW ])[ktsw ];
      f_TSE = (D.f[dirTSE ])[ktse ];
      f_TNW = (D.f[dirTNW ])[ktnw ];
      f_BNE = (D.f[dirBNE ])[kbne ];
      f_BSW = (D.f[dirBSW ])[kbsw ];
      f_BSE = (D.f[dirBSE ])[kbse ];
      f_BNW = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
				f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
				f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W));// / (one + drho); 


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE))  + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                  (f_N - f_S));// / (one + drho); 

      vx3    =    (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                  (f_T - f_B));// / (one + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);// * (one + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
         //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX = c0o1;
      real VeloY = c0o1;
      real VeloZ = c0o1;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  
	  
	  //ToDo anders klammern !!!!!!
	  
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) /** (one + drho)*/-cu_sq); 
         (D.f[dirW])[kw]=((c1o1 - q) * f_E + q * ((f_E + f_W) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloX     )) / (q + c1o1) ;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) /** (one + drho)*/-cu_sq); 
         (D.f[dirE])[ke]=((c1o1 - q) * f_W + q * ((f_W + f_E) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloX     )) / (q + c1o1) ;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirS])[ks]=((c1o1 - q) * f_N + q * ((f_N + f_S) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloY     )) / (q + c1o1) ;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirN])[kn]=((c1o1 - q) * f_S + q * ((f_S + f_N) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloY     )) / (q + c1o1) ;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirB])[kb]=((c1o1 - q) * f_T + q * ((f_T + f_B) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloZ     )) / (q + c1o1) ;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirT])[kt]=((c1o1 - q) * f_B + q * ((f_B + f_T) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloZ     )) / (q + c1o1) ;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirSW])[ksw]=((c1o1 - q) * f_NE + q * ((f_NE + f_SW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX+VeloY)) / (q + c1o1) ;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirNE])[kne]=((c1o1 - q) * f_SW + q * ((f_SW + f_NE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX-VeloY)) / (q + c1o1) ;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirNW])[knw]=((c1o1 - q) * f_SE + q * ((f_SE + f_NW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX-VeloY)) / (q + c1o1) ;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) /** (one + drho)*/-cu_sq); 
         (D.f[dirSE])[kse]=((c1o1 - q) * f_NW + q * ((f_NW + f_SE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX+VeloY)) / (q + c1o1) ;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBW])[kbw]=((c1o1 - q) * f_TE + q * ((f_TE + f_BW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTE])[kte]=((c1o1 - q) * f_BW + q * ((f_BW + f_TE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTW])[ktw]=((c1o1 - q) * f_BE + q * ((f_BE + f_TW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBE])[kbe]=((c1o1 - q) * f_TW + q * ((f_TW + f_BE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBS])[kbs]=((c1o1 - q) * f_TN + q * ((f_TN + f_BS) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTN])[ktn]=((c1o1 - q) * f_BS + q * ((f_BS + f_TN) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTS])[kts]=((c1o1 - q) * f_BN + q * ((f_BN + f_TS) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBN])[kbn]=((c1o1 - q) * f_TS + q * ((f_TS + f_BN) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBSW])[kbsw]=((c1o1 - q) * f_TNE + q * ((f_TNE + f_BSW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX+VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTNE])[ktne]=((c1o1 - q) * f_BSW + q * ((f_BSW + f_TNE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX-VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTSW])[ktsw]=((c1o1 - q) * f_BNE + q * ((f_BNE + f_TSW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX+VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBNE])[kbne]=((c1o1 - q) * f_TSW + q * ((f_TSW + f_BNE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX-VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBNW])[kbnw]=((c1o1 - q) * f_TSE + q * ((f_TSE + f_BNW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX-VeloY+VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTSE])[ktse]=((c1o1 - q) * f_BNW + q * ((f_BNW + f_TSE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX+VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirTNW])[ktnw]=((c1o1 - q) * f_BSE + q * ((f_BSE + f_TNW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX-VeloY-VeloZ)) / (q + c1o1) ;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) /** (one + drho)*/-cu_sq); 
         (D.f[dirBSE])[kbse]=((c1o1 - q) * f_TNW + q * ((f_TNW + f_BSE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX+VeloY+VeloZ)) / (q + c1o1) ;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDeviceCompHighNu27(  int inx,
												 int iny,
												 real* DD, 
												 int* k_Q, 
												 real* QQ,
												 unsigned int sizeQ,
												 int numberOfNodes, 
												 real om1, 
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfNodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = k_Q[k];
      unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
            f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_E   = (D.f[dirE   ])[ke   ];
      f_W   = (D.f[dirW   ])[kw   ];
      f_N   = (D.f[dirN   ])[kn   ];
      f_S   = (D.f[dirS   ])[ks   ];
      f_T   = (D.f[dirT   ])[kt   ];
      f_B   = (D.f[dirB   ])[kb   ];
      f_NE  = (D.f[dirNE  ])[kne  ];
      f_SW  = (D.f[dirSW  ])[ksw  ];
      f_SE  = (D.f[dirSE  ])[kse  ];
      f_NW  = (D.f[dirNW  ])[knw  ];
      f_TE  = (D.f[dirTE  ])[kte  ];
      f_BW  = (D.f[dirBW  ])[kbw  ];
      f_BE  = (D.f[dirBE  ])[kbe  ];
      f_TW  = (D.f[dirTW  ])[ktw  ];
      f_TN  = (D.f[dirTN  ])[ktn  ];
      f_BS  = (D.f[dirBS  ])[kbs  ];
      f_BN  = (D.f[dirBN  ])[kbn  ];
      f_TS  = (D.f[dirTS  ])[kts  ];
      f_TNE = (D.f[dirTNE ])[ktne ];
      f_TSW = (D.f[dirTSW ])[ktsw ];
      f_TSE = (D.f[dirTSE ])[ktse ];
      f_TNW = (D.f[dirTNW ])[ktnw ];
      f_BNE = (D.f[dirBNE ])[kbne ];
      f_BSW = (D.f[dirBSW ])[kbsw ];
      f_BSE = (D.f[dirBSE ])[kbse ];
      f_BNW = (D.f[dirBNW ])[kbnw ];
      //f_W    = (D.f[dirE   ])[ke   ];
      //f_E    = (D.f[dirW   ])[kw   ];
      //f_S    = (D.f[dirN   ])[kn   ];
      //f_N    = (D.f[dirS   ])[ks   ];
      //f_B    = (D.f[dirT   ])[kt   ];
      //f_T    = (D.f[dirB   ])[kb   ];
      //f_SW   = (D.f[dirNE  ])[kne  ];
      //f_NE   = (D.f[dirSW  ])[ksw  ];
      //f_NW   = (D.f[dirSE  ])[kse  ];
      //f_SE   = (D.f[dirNW  ])[knw  ];
      //f_BW   = (D.f[dirTE  ])[kte  ];
      //f_TE   = (D.f[dirBW  ])[kbw  ];
      //f_TW   = (D.f[dirBE  ])[kbe  ];
      //f_BE   = (D.f[dirTW  ])[ktw  ];
      //f_BS   = (D.f[dirTN  ])[ktn  ];
      //f_TN   = (D.f[dirBS  ])[kbs  ];
      //f_TS   = (D.f[dirBN  ])[kbn  ];
      //f_BN   = (D.f[dirTS  ])[kts  ];
      //f_BSW  = (D.f[dirTNE ])[ktne ];
      //f_BNE  = (D.f[dirTSW ])[ktsw ];
      //f_BNW  = (D.f[dirTSE ])[ktse ];
      //f_BSE  = (D.f[dirTNW ])[ktnw ];
      //f_TSW  = (D.f[dirBNE ])[kbne ];
      //f_TNE  = (D.f[dirBSW ])[kbsw ];
      //f_TNW  = (D.f[dirBSE ])[kbse ];
      //f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
				f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
				f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE))  + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                  (f_N - f_S)) / (c1o1 + drho); 

      vx3    =    (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                  (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
         //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX = c0o1;
      real VeloY = c0o1;
      real VeloZ = c0o1;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  
	  
	  //ToDo anders klammern !!!!!!
	  
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=((c1o1 - q) * f_E + q * ((f_E + f_W) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloX     )) / (q + c1o1) ;
         //(D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-f_W+(f_E+f_W-two*feq*om1)/(one-om1))*c1o2+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q) - c2over27 * drho;
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=((c1o1 - q) * f_W + q * ((f_W + f_E) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloX     )) / (q + c1o1) ;
         //(D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-f_E+(f_W+f_E-two*feq*om1)/(one-om1))*c1o2+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q) - c2over27 * drho;
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=((c1o1 - q) * f_N + q * ((f_N + f_S) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloY     )) / (q + c1o1) ;
         //(D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-f_S+(f_N+f_S-two*feq*om1)/(one-om1))*c1o2+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q) - c2over27 * drho;
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=((c1o1 - q) * f_S + q * ((f_S + f_N) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloY     )) / (q + c1o1) ;
         //(D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-f_N+(f_S+f_N-two*feq*om1)/(one-om1))*c1o2+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q) - c2over27 * drho;
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=((c1o1 - q) * f_T + q * ((f_T + f_B) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*( VeloZ     )) / (q + c1o1) ;
         //(D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-f_B+(f_T+f_B-two*feq*om1)/(one-om1))*c1o2+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q) - c2over27 * drho;
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=((c1o1 - q) * f_B + q * ((f_B + f_T) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c2o27*(-VeloZ     )) / (q + c1o1) ;
         //(D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-f_T+(f_B+f_T-two*feq*om1)/(one-om1))*c1o2+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q) - c2over27 * drho;
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=((c1o1 - q) * f_NE + q * ((f_NE + f_SW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX+VeloY)) / (q + c1o1) ;
         //(D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-f_SW+(f_NE+f_SW-two*feq*om1)/(one-om1))*c1o2+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q) - c1over54 * drho;
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=((c1o1 - q) * f_SW + q * ((f_SW + f_NE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX-VeloY)) / (q + c1o1) ;
         //(D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-f_NE+(f_SW+f_NE-two*feq*om1)/(one-om1))*c1o2+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q) - c1over54 * drho;
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=((c1o1 - q) * f_SE + q * ((f_SE + f_NW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX-VeloY)) / (q + c1o1) ;
         //(D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-f_NW+(f_SE+f_NW-two*feq*om1)/(one-om1))*c1o2+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q) - c1over54 * drho;
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=((c1o1 - q) * f_NW + q * ((f_NW + f_SE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX+VeloY)) / (q + c1o1) ;
         //(D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-f_SE+(f_NW+f_SE-two*feq*om1)/(one-om1))*c1o2+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q) - c1over54 * drho;
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=((c1o1 - q) * f_TE + q * ((f_TE + f_BW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-f_BW+(f_TE+f_BW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=((c1o1 - q) * f_BW + q * ((f_BW + f_TE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-f_TE+(f_BW+f_TE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=((c1o1 - q) * f_BE + q * ((f_BE + f_TW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloX-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-f_TW+(f_BE+f_TW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=((c1o1 - q) * f_TW + q * ((f_TW + f_BE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloX+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-f_BE+(f_TW+f_BE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=((c1o1 - q) * f_TN + q * ((f_TN + f_BS) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-f_BS+(f_TN+f_BS-two*feq*om1)/(one-om1))*c1o2+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=((c1o1 - q) * f_BS + q * ((f_BS + f_TN) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-f_TN+(f_BS+f_TN-two*feq*om1)/(one-om1))*c1o2+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=((c1o1 - q) * f_BN + q * ((f_BN + f_TS) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*( VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-f_TS+(f_BN+f_TS-two*feq*om1)/(one-om1))*c1o2+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=((c1o1 - q) * f_TS + q * ((f_TS + f_BN) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o54*(-VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-f_BN+(f_TS+f_BN-two*feq*om1)/(one-om1))*c1o2+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q) - c1over54 * drho;
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=((c1o1 - q) * f_TNE + q * ((f_TNE + f_BSW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX+VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=((c1o1 - q) * f_BSW + q * ((f_BSW + f_TNE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX-VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=((c1o1 - q) * f_BNE + q * ((f_BNE + f_TSW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX+VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=((c1o1 - q) * f_TSW + q * ((f_TSW + f_BNE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX-VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=((c1o1 - q) * f_TSE + q * ((f_TSE + f_BNW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX-VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=((c1o1 - q) * f_BNW + q * ((f_BNW + f_TSE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX+VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=((c1o1 - q) * f_BSE + q * ((f_BSE + f_TNW) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*( VeloX-VeloY-VeloZ)) / (q + c1o1) ;
         //(D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=((c1o1 - q) * f_TNW + q * ((f_TNW + f_BSE) * (c1o1 - om1) + om1 * c2o1 * feq) - c6o1*c1o216*(-VeloX+VeloY+VeloZ)) / (q + c1o1) ;
         //(D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDeviceComp27(int inx,
										 int iny,
										 real* DD, 
										 int* k_Q, 
										 real* QQ,
										 unsigned int numberOfBCnodes, 
										 real om1, 
										 unsigned int* neighborX,
										 unsigned int* neighborY,
										 unsigned int* neighborZ,
										 unsigned int size_Mat, 
										 bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k < numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   * numberOfBCnodes];
      q_dirW   = &QQ[dirW   * numberOfBCnodes];
      q_dirN   = &QQ[dirN   * numberOfBCnodes];
      q_dirS   = &QQ[dirS   * numberOfBCnodes];
      q_dirT   = &QQ[dirT   * numberOfBCnodes];
      q_dirB   = &QQ[dirB   * numberOfBCnodes];
      q_dirNE  = &QQ[dirNE  * numberOfBCnodes];
      q_dirSW  = &QQ[dirSW  * numberOfBCnodes];
      q_dirSE  = &QQ[dirSE  * numberOfBCnodes];
      q_dirNW  = &QQ[dirNW  * numberOfBCnodes];
      q_dirTE  = &QQ[dirTE  * numberOfBCnodes];
      q_dirBW  = &QQ[dirBW  * numberOfBCnodes];
      q_dirBE  = &QQ[dirBE  * numberOfBCnodes];
      q_dirTW  = &QQ[dirTW  * numberOfBCnodes];
      q_dirTN  = &QQ[dirTN  * numberOfBCnodes];
      q_dirBS  = &QQ[dirBS  * numberOfBCnodes];
      q_dirBN  = &QQ[dirBN  * numberOfBCnodes];
      q_dirTS  = &QQ[dirTS  * numberOfBCnodes];
      q_dirTNE = &QQ[dirTNE * numberOfBCnodes];
      q_dirTSW = &QQ[dirTSW * numberOfBCnodes];
      q_dirTSE = &QQ[dirTSE * numberOfBCnodes];
      q_dirTNW = &QQ[dirTNW * numberOfBCnodes];
      q_dirBNE = &QQ[dirBNE * numberOfBCnodes];
      q_dirBSW = &QQ[dirBSW * numberOfBCnodes];
      q_dirBSE = &QQ[dirBSE * numberOfBCnodes];
      q_dirBNW = &QQ[dirBNW * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = k_Q[k];
      unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
            f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dirE   ])[ke   ];
      f_E    = (D.f[dirW   ])[kw   ];
      f_S    = (D.f[dirN   ])[kn   ];
      f_N    = (D.f[dirS   ])[ks   ];
      f_B    = (D.f[dirT   ])[kt   ];
      f_T    = (D.f[dirB   ])[kb   ];
      f_SW   = (D.f[dirNE  ])[kne  ];
      f_NE   = (D.f[dirSW  ])[ksw  ];
      f_NW   = (D.f[dirSE  ])[kse  ];
      f_SE   = (D.f[dirNW  ])[knw  ];
      f_BW   = (D.f[dirTE  ])[kte  ];
      f_TE   = (D.f[dirBW  ])[kbw  ];
      f_TW   = (D.f[dirBE  ])[kbe  ];
      f_BE   = (D.f[dirTW  ])[ktw  ];
      f_BS   = (D.f[dirTN  ])[ktn  ];
      f_TN   = (D.f[dirBS  ])[kbs  ];
      f_TS   = (D.f[dirBN  ])[kbn  ];
      f_BN   = (D.f[dirTS  ])[kts  ];
      f_BSW  = (D.f[dirTNE ])[ktne ];
      f_BNE  = (D.f[dirTSW ])[ktsw ];
      f_BNW  = (D.f[dirTSE ])[ktse ];
      f_BSE  = (D.f[dirTNW ])[ktnw ];
      f_TSW  = (D.f[dirBNE ])[kbne ];
      f_TNE  = (D.f[dirBSW ])[kbsw ];
      f_TNW  = (D.f[dirBSE ])[kbse ];
      f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
				f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
				f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE))  + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                  (f_N - f_S)) / (c1o1 + drho); 

      vx3    =    (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                  (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
         //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  
	  
	  //ToDo anders klammern !!!!!!
	  
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W))/(c1o1+q);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E))/(c1o1+q);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S))/(c1o1+q);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N))/(c1o1+q);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B))/(c1o1+q);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T))/(c1o1+q);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW))/(c1o1+q);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE))/(c1o1+q);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW))/(c1o1+q);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE))/(c1o1+q);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW))/(c1o1+q);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE))/(c1o1+q);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW))/(c1o1+q);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE))/(c1o1+q);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS))/(c1o1+q);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN))/(c1o1+q);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS))/(c1o1+q);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN))/(c1o1+q);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q);
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDevice27(int inx,
                                     int iny,
                                     real* DD, 
                                     int* k_Q, 
                                     real* QQ,
                                     unsigned int numberOfBCnodes, 
                                     real om1, 
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat, 
                                     bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *numberOfBCnodes];
      q_dirW   = &QQ[dirW   *numberOfBCnodes];
      q_dirN   = &QQ[dirN   *numberOfBCnodes];
      q_dirS   = &QQ[dirS   *numberOfBCnodes];
      q_dirT   = &QQ[dirT   *numberOfBCnodes];
      q_dirB   = &QQ[dirB   *numberOfBCnodes];
      q_dirNE  = &QQ[dirNE  *numberOfBCnodes];
      q_dirSW  = &QQ[dirSW  *numberOfBCnodes];
      q_dirSE  = &QQ[dirSE  *numberOfBCnodes];
      q_dirNW  = &QQ[dirNW  *numberOfBCnodes];
      q_dirTE  = &QQ[dirTE  *numberOfBCnodes];
      q_dirBW  = &QQ[dirBW  *numberOfBCnodes];
      q_dirBE  = &QQ[dirBE  *numberOfBCnodes];
      q_dirTW  = &QQ[dirTW  *numberOfBCnodes];
      q_dirTN  = &QQ[dirTN  *numberOfBCnodes];
      q_dirBS  = &QQ[dirBS  *numberOfBCnodes];
      q_dirBN  = &QQ[dirBN  *numberOfBCnodes];
      q_dirTS  = &QQ[dirTS  *numberOfBCnodes];
      q_dirTNE = &QQ[dirTNE *numberOfBCnodes];
      q_dirTSW = &QQ[dirTSW *numberOfBCnodes];
      q_dirTSE = &QQ[dirTSE *numberOfBCnodes];
      q_dirTNW = &QQ[dirTNW *numberOfBCnodes];
      q_dirBNE = &QQ[dirBNE *numberOfBCnodes];
      q_dirBSW = &QQ[dirBSW *numberOfBCnodes];
      q_dirBSE = &QQ[dirBSE *numberOfBCnodes];
      q_dirBNW = &QQ[dirBNW *numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = k_Q[k];
      unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      //unsigned int nxny = nx*ny;
      //unsigned int kzero= numberOfNodesK;
      //unsigned int ke   = numberOfNodesK;
      //unsigned int kw   = numberOfNodesK + 1;
      //unsigned int kn   = numberOfNodesK;
      //unsigned int ks   = numberOfNodesK + nx;
      //unsigned int kt   = numberOfNodesK;
      //unsigned int kb   = numberOfNodesK + nxny;
      //unsigned int ksw  = numberOfNodesK + nx + 1;
      //unsigned int kne  = numberOfNodesK;
      //unsigned int kse  = numberOfNodesK + nx;
      //unsigned int knw  = numberOfNodesK + 1;
      //unsigned int kbw  = numberOfNodesK + nxny + 1;
      //unsigned int kte  = numberOfNodesK;
      //unsigned int kbe  = numberOfNodesK + nxny;
      //unsigned int ktw  = numberOfNodesK + 1;
      //unsigned int kbs  = numberOfNodesK + nxny + nx;
      //unsigned int ktn  = numberOfNodesK;
      //unsigned int kbn  = numberOfNodesK + nxny;
      //unsigned int kts  = numberOfNodesK + nx;
      //unsigned int ktse = numberOfNodesK + nx;
      //unsigned int kbnw = numberOfNodesK + nxny + 1;
      //unsigned int ktnw = numberOfNodesK + 1;
      //unsigned int kbse = numberOfNodesK + nxny + nx;
      //unsigned int ktsw = numberOfNodesK + nx + 1;
      //unsigned int kbne = numberOfNodesK + nxny;
      //unsigned int ktne = numberOfNodesK;
      //unsigned int kbsw = numberOfNodesK + nxny + nx + 1;
      ////////////////////////////////////////////////////////////////////////////////
      //real vx1, vx2, vx3, drho, feq, q;
      //drho    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
      //            (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
      //            (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
      //            (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
      //            (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
      //            (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
      //            (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
      //            (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
      //            (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
      //            (D.f[dirZERO])[kzero]+ 
      //            (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
      //            (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
      //            (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
      //            (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

      //vx1    =    (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
      //            (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
      //            (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
      //            (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
      //            (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
      //            (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
      //            (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
      //            (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
      //            (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

      //vx2    =    (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
      //            (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
      //            (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
      //            (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
      //            (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
      //            (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
      //            (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
      //            (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
      //            (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

      //vx3    =    (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
      //            (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
      //            (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
      //            (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
      //            (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
      //            (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
      //            (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
      //            (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
      //            (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

      //real cu_sq=1.5f*(vx1*vx1+vx2*vx2+vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
            f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dirE   ])[ke   ];
      f_E    = (D.f[dirW   ])[kw   ];
      f_S    = (D.f[dirN   ])[kn   ];
      f_N    = (D.f[dirS   ])[ks   ];
      f_B    = (D.f[dirT   ])[kt   ];
      f_T    = (D.f[dirB   ])[kb   ];
      f_SW   = (D.f[dirNE  ])[kne  ];
      f_NE   = (D.f[dirSW  ])[ksw  ];
      f_NW   = (D.f[dirSE  ])[kse  ];
      f_SE   = (D.f[dirNW  ])[knw  ];
      f_BW   = (D.f[dirTE  ])[kte  ];
      f_TE   = (D.f[dirBW  ])[kbw  ];
      f_TW   = (D.f[dirBE  ])[kbe  ];
      f_BE   = (D.f[dirTW  ])[ktw  ];
      f_BS   = (D.f[dirTN  ])[ktn  ];
      f_TN   = (D.f[dirBS  ])[kbs  ];
      f_TS   = (D.f[dirBN  ])[kbn  ];
      f_BN   = (D.f[dirTS  ])[kts  ];
      f_BSW  = (D.f[dirTNE ])[ktne ];
      f_BNE  = (D.f[dirTSW ])[ktsw ];
      f_BNW  = (D.f[dirTSE ])[ktse ];
      f_BSE  = (D.f[dirTNW ])[ktnw ];
      f_TSW  = (D.f[dirBNE ])[kbne ];
      f_TNE  = (D.f[dirBSW ])[kbsw ];
      f_TNW  = (D.f[dirBSE ])[kbse ];
      f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
				f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
				f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  ((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W); 


      vx2    =   (-(f_TSE - f_BNW) + (f_TNW - f_BSE))  + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                  (f_N - f_S); 

      vx3    =    ((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                  (f_T - f_B); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

	  //b�ser lecktest
	  //q = q_dirE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirW])[kw]=999.f;
   //   }

   //   q = q_dirW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirE])[ke]=999.f;
   //   }

   //   q = q_dirN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirS])[ks]=999.f;
   //   }

   //   q = q_dirS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirN])[kn]=999.f;
   //   }

   //   q = q_dirT[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirB])[kb]=999.f;
   //   }

   //   q = q_dirB[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirT])[kt]=999.f;
   //   }

   //   q = q_dirNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirSW])[ksw]=999.f;
   //   }

   //   q = q_dirSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirNE])[kne]=999.f;
   //   }

   //   q = q_dirSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirNW])[knw]=999.f;
   //   }

   //   q = q_dirNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirSE])[kse]=999.f;
   //   }

   //   q = q_dirTE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirBW])[kbw]=999.f;
   //   }

   //   q = q_dirBW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirTE])[kte]=999.f;
   //   }

   //   q = q_dirBE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirTW])[ktw]=999.f;
   //   }

   //   q = q_dirTW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirBE])[kbe]=999.f;
   //   }

   //   q = q_dirTN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirBS])[kbs]=999.f;
   //   }

   //   q = q_dirBS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirTN])[ktn]=999.f;
   //   }

   //   q = q_dirBN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirTS])[kts]=999.f;
   //   }

   //   q = q_dirTS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirBN])[kbn]=999.f;
   //   }

   //   q = q_dirTNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirBSW])[kbsw]=999.f;
   //   }

   //   q = q_dirBSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirTNE])[ktne]=999.f;
   //   }

   //   q = q_dirBNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirTSW])[ktsw]=999.f;
   //   }

   //   q = q_dirTSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirBNE])[kbne]=999.f;
   //   }

   //   q = q_dirTSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirBNW])[kbnw]=999.f;
   //   }

   //   q = q_dirBNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirTSE])[ktse]=999.f;
   //   }

   //   q = q_dirBSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirTNW])[ktnw]=999.f;
   //   }

   //   q = q_dirTNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      (D.f[dirBSE])[kbse]=999.f;
   //   }

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
         //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  
	  
	  //ToDo anders klammern !!!!!!
	  
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        )-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W))/(c1o1+q);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        )-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E))/(c1o1+q);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    )-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S))/(c1o1+q);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    )-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N))/(c1o1+q);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B))/(c1o1+q);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T))/(c1o1+q);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW))/(c1o1+q);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE))/(c1o1+q);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW))/(c1o1+q);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE))/(c1o1+q);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW))/(c1o1+q);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE))/(c1o1+q);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW))/(c1o1+q);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE))/(c1o1+q);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS))/(c1o1+q);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN))/(c1o1+q);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS))/(c1o1+q);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN))/(c1o1+q);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q);
         //(D.f[dirBSE])[kbse]=zero;
      }

	 // q = q_dirE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cu_sq); 
  //       (D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-feq*om1)/(one-om1)+q/(one+q)*(f_E+f_W);
		//// (D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-f_W+(f_E+f_W-two*feq*om1)/(one-om1))*c1o2+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);
  //    }

  //    q = q_dirW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cu_sq); 
  //       (D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-feq*om1)/(one-om1)+q/(one+q)*(f_W+f_E);
  //    }

  //    q = q_dirN[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cu_sq); 
  //       (D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-feq*om1)/(one-om1)+q/(one+q)*(f_N+f_S);
  //    }

  //    q = q_dirS[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cu_sq); 
  //       (D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-feq*om1)/(one-om1)+q/(one+q)*(f_S+f_N);
  //    }

  //    q = q_dirT[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cu_sq); 
  //       (D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-feq*om1)/(one-om1)+q/(one+q)*(f_T+f_B);
  //    }

  //    q = q_dirB[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cu_sq); 
  //       (D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-feq*om1)/(one-om1)+q/(one+q)*(f_B+f_T);
  //    }

  //    q = q_dirNE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
  //       (D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-feq*om1)/(one-om1)+q/(one+q)*(f_NE+f_SW);
  //    }

  //    q = q_dirSW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
  //       (D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-feq*om1)/(one-om1)+q/(one+q)*(f_SW+f_NE);
  //    }

  //    q = q_dirSE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
  //       (D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-feq*om1)/(one-om1)+q/(one+q)*(f_SE+f_NW);
  //    }

  //    q = q_dirNW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
  //       (D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-feq*om1)/(one-om1)+q/(one+q)*(f_NW+f_SE);
  //    }

  //    q = q_dirTE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
  //       (D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-feq*om1)/(one-om1)+q/(one+q)*(f_TE+f_BW);
  //    }

  //    q = q_dirBW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
  //       (D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-feq*om1)/(one-om1)+q/(one+q)*(f_BW+f_TE);
  //    }

  //    q = q_dirBE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
  //       (D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-feq*om1)/(one-om1)+q/(one+q)*(f_BE+f_TW);
  //    }

  //    q = q_dirTW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
  //       (D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-feq*om1)/(one-om1)+q/(one+q)*(f_TW+f_BE);
  //    }

  //    q = q_dirTN[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
  //       (D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-feq*om1)/(one-om1)+q/(one+q)*(f_TN+f_BS);
  //    }

  //    q = q_dirBS[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
  //       (D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-feq*om1)/(one-om1)+q/(one+q)*(f_BS+f_TN);
  //    }

  //    q = q_dirBN[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
  //       (D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-feq*om1)/(one-om1)+q/(one+q)*(f_BN+f_TS);
  //    }

  //    q = q_dirTS[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
  //       (D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-feq*om1)/(one-om1)+q/(one+q)*(f_TS+f_BN);
  //    }

  //    q = q_dirTNE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
  //       (D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-feq*om1)/(one-om1)+q/(one+q)*(f_TNE+f_BSW);
  //    }

  //    q = q_dirBSW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
  //       (D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-feq*om1)/(one-om1)+q/(one+q)*(f_BSW+f_TNE);
  //    }

  //    q = q_dirBNE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
  //       (D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-feq*om1)/(one-om1)+q/(one+q)*(f_BNE+f_TSW);
  //    }

  //    q = q_dirTSW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
  //       (D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-feq*om1)/(one-om1)+q/(one+q)*(f_TSW+f_BNE);
  //    }

  //    q = q_dirTSE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
  //       (D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-feq*om1)/(one-om1)+q/(one+q)*(f_TSE+f_BNW);
  //    }

  //    q = q_dirBNW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
  //       (D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-feq*om1)/(one-om1)+q/(one+q)*(f_BNW+f_TSE);
  //    }

  //    q = q_dirBSE[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
  //       (D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-feq*om1)/(one-om1)+q/(one+q)*(f_BSE+f_TNW);
  //    }

  //    q = q_dirTNW[k];
  //    if (q>=zero && q<=one)
  //    {
  //       feq=c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
  //       (D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-feq*om1)/(one-om1)+q/(one+q)*(f_TNW+f_BSE);
  //    }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void BBDevice27(int inx,
                                     int iny,
                                     real* DD, 
                                     int* k_Q, 
                                     real* QQ,
                                     unsigned int numberOfBCnodes, 
                                     real om1, 
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     unsigned int size_Mat, 
                                     bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
         *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
         *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
         *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
         *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *numberOfBCnodes];
      q_dirW   = &QQ[dirW   *numberOfBCnodes];
      q_dirN   = &QQ[dirN   *numberOfBCnodes];
      q_dirS   = &QQ[dirS   *numberOfBCnodes];
      q_dirT   = &QQ[dirT   *numberOfBCnodes];
      q_dirB   = &QQ[dirB   *numberOfBCnodes];
      q_dirNE  = &QQ[dirNE  *numberOfBCnodes];
      q_dirSW  = &QQ[dirSW  *numberOfBCnodes];
      q_dirSE  = &QQ[dirSE  *numberOfBCnodes];
      q_dirNW  = &QQ[dirNW  *numberOfBCnodes];
      q_dirTE  = &QQ[dirTE  *numberOfBCnodes];
      q_dirBW  = &QQ[dirBW  *numberOfBCnodes];
      q_dirBE  = &QQ[dirBE  *numberOfBCnodes];
      q_dirTW  = &QQ[dirTW  *numberOfBCnodes];
      q_dirTN  = &QQ[dirTN  *numberOfBCnodes];
      q_dirBS  = &QQ[dirBS  *numberOfBCnodes];
      q_dirBN  = &QQ[dirBN  *numberOfBCnodes];
      q_dirTS  = &QQ[dirTS  *numberOfBCnodes];
      q_dirTNE = &QQ[dirTNE *numberOfBCnodes];
      q_dirTSW = &QQ[dirTSW *numberOfBCnodes];
      q_dirTSE = &QQ[dirTSE *numberOfBCnodes];
      q_dirTNW = &QQ[dirTNW *numberOfBCnodes];
      q_dirBNE = &QQ[dirBNE *numberOfBCnodes];
      q_dirBSW = &QQ[dirBSW *numberOfBCnodes];
      q_dirBSE = &QQ[dirBSE *numberOfBCnodes];
      q_dirBNW = &QQ[dirBNW *numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int numberOfNodesK  = k_Q[k];
      //unsigned int kzero= numberOfNodesK;
      unsigned int ke   = numberOfNodesK;
      unsigned int kw   = neighborX[numberOfNodesK];
      unsigned int kn   = numberOfNodesK;
      unsigned int ks   = neighborY[numberOfNodesK];
      unsigned int kt   = numberOfNodesK;
      unsigned int kb   = neighborZ[numberOfNodesK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = numberOfNodesK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = numberOfNodesK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = numberOfNodesK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = numberOfNodesK;
      unsigned int kbsw = neighborZ[ksw];
      //unsigned int nxny = nx*ny;
      //unsigned int kzero= numberOfNodesK;
      //unsigned int ke   = numberOfNodesK;
      //unsigned int kw   = numberOfNodesK + 1;
      //unsigned int kn   = numberOfNodesK;
      //unsigned int ks   = numberOfNodesK + nx;
      //unsigned int kt   = numberOfNodesK;
      //unsigned int kb   = numberOfNodesK + nxny;
      //unsigned int ksw  = numberOfNodesK + nx + 1;
      //unsigned int kne  = numberOfNodesK;
      //unsigned int kse  = numberOfNodesK + nx;
      //unsigned int knw  = numberOfNodesK + 1;
      //unsigned int kbw  = numberOfNodesK + nxny + 1;
      //unsigned int kte  = numberOfNodesK;
      //unsigned int kbe  = numberOfNodesK + nxny;
      //unsigned int ktw  = numberOfNodesK + 1;
      //unsigned int kbs  = numberOfNodesK + nxny + nx;
      //unsigned int ktn  = numberOfNodesK;
      //unsigned int kbn  = numberOfNodesK + nxny;
      //unsigned int kts  = numberOfNodesK + nx;
      //unsigned int ktse = numberOfNodesK + nx;
      //unsigned int kbnw = numberOfNodesK + nxny + 1;
      //unsigned int ktnw = numberOfNodesK + 1;
      //unsigned int kbse = numberOfNodesK + nxny + nx;
      //unsigned int ktsw = numberOfNodesK + nx + 1;
      //unsigned int kbne = numberOfNodesK + nxny;
      //unsigned int ktne = numberOfNodesK;
      //unsigned int kbsw = numberOfNodesK + nxny + nx + 1;
      ////////////////////////////////////////////////////////////////////////////////
     
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dirE   ])[ke   ];
      f_E    = (D.f[dirW   ])[kw   ];
      f_S    = (D.f[dirN   ])[kn   ];
      f_N    = (D.f[dirS   ])[ks   ];
      f_B    = (D.f[dirT   ])[kt   ];
      f_T    = (D.f[dirB   ])[kb   ];
      f_SW   = (D.f[dirNE  ])[kne  ];
      f_NE   = (D.f[dirSW  ])[ksw  ];
      f_NW   = (D.f[dirSE  ])[kse  ];
      f_SE   = (D.f[dirNW  ])[knw  ];
      f_BW   = (D.f[dirTE  ])[kte  ];
      f_TE   = (D.f[dirBW  ])[kbw  ];
      f_TW   = (D.f[dirBE  ])[kbe  ];
      f_BE   = (D.f[dirTW  ])[ktw  ];
      f_BS   = (D.f[dirTN  ])[ktn  ];
      f_TN   = (D.f[dirBS  ])[kbs  ];
      f_TS   = (D.f[dirBN  ])[kbn  ];
      f_BN   = (D.f[dirTS  ])[kts  ];
      f_BSW  = (D.f[dirTNE ])[ktne ];
      f_BNE  = (D.f[dirTSW ])[ktsw ];
      f_BNW  = (D.f[dirTSE ])[ktse ];
      f_BSE  = (D.f[dirTNW ])[ktnw ];
      f_TSW  = (D.f[dirBNE ])[kbne ];
      f_TNE  = (D.f[dirBSW ])[kbsw ];
      f_TNW  = (D.f[dirBSE ])[kbse ];
      f_TSE  = (D.f[dirBNW ])[kbnw ];

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real q;
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirW])[kw]=f_E;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirE])[ke]=f_W;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirS])[ks]=f_N;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirN])[kn]=f_S;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirB])[kb]=f_T;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirT])[kt]=f_B;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirSW])[ksw]=f_NE;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirNE])[kne]=f_SW;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirNW])[knw]=f_SE;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirSE])[kse]=f_NW;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirBW])[kbw]=f_TE;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirTE])[kte]=f_BW;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirTW])[ktw]=f_BE;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirBE])[kbe]=f_TW;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirBS])[kbs]=f_TN;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirTN])[ktn]=f_BS;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirTS])[kts]=f_BN;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirBN])[kbn]=f_TS;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirBSW])[kbsw]=f_TNE;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirTNE])[ktne]=f_BSW;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirTSW])[ktsw]=f_BNE;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirBNE])[kbne]=f_TSW;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirBNW])[kbnw]=f_TSE;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirTSE])[ktse]=f_BNW;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirTNW])[ktnw]=f_BSE;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dirBSE])[kbse]=f_TNW;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

