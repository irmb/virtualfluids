#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __         
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |        
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |        
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |        
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____    
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|   
//      \    \  |    |   ________________________________________________________________    
//       \    \ |    |  |  ______________________________________________________________|   
//        \    \|    |  |  |         __          __     __     __     ______      _______    
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)   
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______    
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/   
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can 
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of 
//  the License, or (at your option) any later version.
//  
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT 
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or 
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License 
//  for more details.
//  
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file VelocityBCs27.cu
//! \ingroup GPU
//! \author Martin Schoenherr
//=======================================================================================
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include "Core/RealConstants.h"

//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QVelDevPlainBB27(
	real* vx,
	real* vy,
	real* vz,
	real* distributions,
	int* k_Q, 
	real* QQ,
	uint sizeQ,
	int kQ, 
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	uint size_Mat, 
	bool isEvenTimestep)
{
	//////////////////////////////////////////////////////////////////////////
	//! The velocity boundary condition is executed in the following steps
	//!
	////////////////////////////////////////////////////////////////////////////////
	//! - Get node index coordinates from thredIdx, blockIdx, blockDim and gridDim.
	//!
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;

   //////////////////////////////////////////////////////////////////////////
   // run for all indices in size of boundary condition (kQ)
   if(k<kQ)
   {
	   //////////////////////////////////////////////////////////////////////////
	   //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
	   //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
	   //!
	   Distributions27 dist;
	   if (isEvenTimestep)
	   {
		  dist.f[dirE   ] = &distributions[dirE   *size_Mat];
		  dist.f[dirW   ] = &distributions[dirW   *size_Mat];
		  dist.f[dirN   ] = &distributions[dirN   *size_Mat];
		  dist.f[dirS   ] = &distributions[dirS   *size_Mat];
		  dist.f[dirT   ] = &distributions[dirT   *size_Mat];
		  dist.f[dirB   ] = &distributions[dirB   *size_Mat];
		  dist.f[dirNE  ] = &distributions[dirNE  *size_Mat];
		  dist.f[dirSW  ] = &distributions[dirSW  *size_Mat];
		  dist.f[dirSE  ] = &distributions[dirSE  *size_Mat];
		  dist.f[dirNW  ] = &distributions[dirNW  *size_Mat];
		  dist.f[dirTE  ] = &distributions[dirTE  *size_Mat];
		  dist.f[dirBW  ] = &distributions[dirBW  *size_Mat];
		  dist.f[dirBE  ] = &distributions[dirBE  *size_Mat];
		  dist.f[dirTW  ] = &distributions[dirTW  *size_Mat];
		  dist.f[dirTN  ] = &distributions[dirTN  *size_Mat];
		  dist.f[dirBS  ] = &distributions[dirBS  *size_Mat];
		  dist.f[dirBN  ] = &distributions[dirBN  *size_Mat];
		  dist.f[dirTS  ] = &distributions[dirTS  *size_Mat];
		  dist.f[dirREST] = &distributions[dirREST*size_Mat];
		  dist.f[dirTNE ] = &distributions[dirTNE *size_Mat];
		  dist.f[dirTSW ] = &distributions[dirTSW *size_Mat];
		  dist.f[dirTSE ] = &distributions[dirTSE *size_Mat];
		  dist.f[dirTNW ] = &distributions[dirTNW *size_Mat];
		  dist.f[dirBNE ] = &distributions[dirBNE *size_Mat];
		  dist.f[dirBSW ] = &distributions[dirBSW *size_Mat];
		  dist.f[dirBSE ] = &distributions[dirBSE *size_Mat];
		  dist.f[dirBNW ] = &distributions[dirBNW *size_Mat];
	   } 
	   else
	   {
		  dist.f[dirW   ] = &distributions[dirE   *size_Mat];
		  dist.f[dirE   ] = &distributions[dirW   *size_Mat];
		  dist.f[dirS   ] = &distributions[dirN   *size_Mat];
		  dist.f[dirN   ] = &distributions[dirS   *size_Mat];
		  dist.f[dirB   ] = &distributions[dirT   *size_Mat];
		  dist.f[dirT   ] = &distributions[dirB   *size_Mat];
		  dist.f[dirSW  ] = &distributions[dirNE  *size_Mat];
		  dist.f[dirNE  ] = &distributions[dirSW  *size_Mat];
		  dist.f[dirNW  ] = &distributions[dirSE  *size_Mat];
		  dist.f[dirSE  ] = &distributions[dirNW  *size_Mat];
		  dist.f[dirBW  ] = &distributions[dirTE  *size_Mat];
		  dist.f[dirTE  ] = &distributions[dirBW  *size_Mat];
		  dist.f[dirTW  ] = &distributions[dirBE  *size_Mat];
		  dist.f[dirBE  ] = &distributions[dirTW  *size_Mat];
		  dist.f[dirBS  ] = &distributions[dirTN  *size_Mat];
		  dist.f[dirTN  ] = &distributions[dirBS  *size_Mat];
		  dist.f[dirTS  ] = &distributions[dirBN  *size_Mat];
		  dist.f[dirBN  ] = &distributions[dirTS  *size_Mat];
		  dist.f[dirREST] = &distributions[dirREST*size_Mat];
		  dist.f[dirTNE ] = &distributions[dirBSW *size_Mat];
		  dist.f[dirTSW ] = &distributions[dirBNE *size_Mat];
		  dist.f[dirTSE ] = &distributions[dirBNW *size_Mat];
		  dist.f[dirTNW ] = &distributions[dirBSE *size_Mat];
		  dist.f[dirBNE ] = &distributions[dirTSW *size_Mat];
		  dist.f[dirBSW ] = &distributions[dirTNE *size_Mat];
		  dist.f[dirBSE ] = &distributions[dirTNW *size_Mat];
		  dist.f[dirBNW ] = &distributions[dirTSE *size_Mat];
	   }


	  ////////////////////////////////////////////////////////////////////////////////
	  //! - Set local velocities
	  //!
	  real VeloX = vx[k];
	  real VeloY = vy[k];
	  real VeloZ = vz[k];
      ////////////////////////////////////////////////////////////////////////////////
	  //! - Set local subgrid distances (q's)
	  //!
      real   *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
			 *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
			 *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
			 *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
			 *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
	  //! - Set neighbor indices (necessary for indirect addressing)  
	  //!
	  uint KQK = k_Q[k];
      uint ke   = KQK;
      uint kw   = neighborX[KQK];
      uint kn   = KQK;
      uint ks   = neighborY[KQK];
      uint kt   = KQK;
      uint kb   = neighborZ[KQK];
      uint ksw  = neighborY[kw];
      uint kne  = KQK;
      uint kse  = ks;
      uint knw  = kw;
      uint kbw  = neighborZ[kw];
      uint kte  = KQK;
      uint kbe  = kb;
      uint ktw  = kw;
      uint kbs  = neighborZ[ks];
      uint ktn  = KQK;
      uint kbn  = kb;
      uint kts  = ks;
      uint ktse = ks;
      uint kbnw = kbw;
      uint ktnw = kw;
      uint kbse = kbs;
      uint ktsw = ksw;
      uint kbne = kb;
      uint ktne = KQK;
      uint kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
     
      ////////////////////////////////////////////////////////////////////////////////
	  //! - Set local distributions  
	  //!
	  real f_W = (dist.f[dirE])[ke];
      real f_E    = (dist.f[dirW   ])[kw   ];
      real f_S    = (dist.f[dirN   ])[kn   ];
      real f_N    = (dist.f[dirS   ])[ks   ];
      real f_B    = (dist.f[dirT   ])[kt   ];
      real f_T    = (dist.f[dirB   ])[kb   ];
      real f_SW   = (dist.f[dirNE  ])[kne  ];
      real f_NE   = (dist.f[dirSW  ])[ksw  ];
      real f_NW   = (dist.f[dirSE  ])[kse  ];
      real f_SE   = (dist.f[dirNW  ])[knw  ];
      real f_BW   = (dist.f[dirTE  ])[kte  ];
      real f_TE   = (dist.f[dirBW  ])[kbw  ];
      real f_TW   = (dist.f[dirBE  ])[kbe  ];
      real f_BE   = (dist.f[dirTW  ])[ktw  ];
      real f_BS   = (dist.f[dirTN  ])[ktn  ];
      real f_TN   = (dist.f[dirBS  ])[kbs  ];
      real f_TS   = (dist.f[dirBN  ])[kbn  ];
      real f_BN   = (dist.f[dirTS  ])[kts  ];
      real f_BSW  = (dist.f[dirTNE ])[ktne ];
      real f_BNE  = (dist.f[dirTSW ])[ktsw ];
      real f_BNW  = (dist.f[dirTSE ])[ktse ];
      real f_BSE  = (dist.f[dirTNW ])[ktnw ];
      real f_TSW  = (dist.f[dirBNE ])[kbne ];
      real f_TNE  = (dist.f[dirBSW ])[kbsw ];
      real f_TNW  = (dist.f[dirBSE ])[kbse ];
      real f_TSE  = (dist.f[dirBNW ])[kbnw ];
	  ////////////////////////////////////////////////////////////////////////////////

	  ////////////////////////////////////////////////////////////////////////////////
	  //! - change the pointer to write the results in the correct array  
	  //!
	  if (!isEvenTimestep)
      {
         dist.f[dirE   ] = &distributions[dirE   *size_Mat];
         dist.f[dirW   ] = &distributions[dirW   *size_Mat];
         dist.f[dirN   ] = &distributions[dirN   *size_Mat];
         dist.f[dirS   ] = &distributions[dirS   *size_Mat];
         dist.f[dirT   ] = &distributions[dirT   *size_Mat];
         dist.f[dirB   ] = &distributions[dirB   *size_Mat];
         dist.f[dirNE  ] = &distributions[dirNE  *size_Mat];
         dist.f[dirSW  ] = &distributions[dirSW  *size_Mat];
         dist.f[dirSE  ] = &distributions[dirSE  *size_Mat];
         dist.f[dirNW  ] = &distributions[dirNW  *size_Mat];
         dist.f[dirTE  ] = &distributions[dirTE  *size_Mat];
         dist.f[dirBW  ] = &distributions[dirBW  *size_Mat];
         dist.f[dirBE  ] = &distributions[dirBE  *size_Mat];
         dist.f[dirTW  ] = &distributions[dirTW  *size_Mat];
         dist.f[dirTN  ] = &distributions[dirTN  *size_Mat];
         dist.f[dirBS  ] = &distributions[dirBS  *size_Mat];
         dist.f[dirBN  ] = &distributions[dirBN  *size_Mat];
         dist.f[dirTS  ] = &distributions[dirTS  *size_Mat];
         dist.f[dirREST] = &distributions[dirREST*size_Mat];
         dist.f[dirTNE ] = &distributions[dirTNE *size_Mat];
         dist.f[dirTSW ] = &distributions[dirTSW *size_Mat];
         dist.f[dirTSE ] = &distributions[dirTSE *size_Mat];
         dist.f[dirTNW ] = &distributions[dirTNW *size_Mat];
         dist.f[dirBNE ] = &distributions[dirBNE *size_Mat];
         dist.f[dirBSW ] = &distributions[dirBSW *size_Mat];
         dist.f[dirBSE ] = &distributions[dirBSE *size_Mat];
         dist.f[dirBNW ] = &distributions[dirBNW *size_Mat];
      } 
      else
      {
         dist.f[dirW   ] = &distributions[dirE   *size_Mat];
         dist.f[dirE   ] = &distributions[dirW   *size_Mat];
         dist.f[dirS   ] = &distributions[dirN   *size_Mat];
         dist.f[dirN   ] = &distributions[dirS   *size_Mat];
         dist.f[dirB   ] = &distributions[dirT   *size_Mat];
         dist.f[dirT   ] = &distributions[dirB   *size_Mat];
         dist.f[dirSW  ] = &distributions[dirNE  *size_Mat];
         dist.f[dirNE  ] = &distributions[dirSW  *size_Mat];
         dist.f[dirNW  ] = &distributions[dirSE  *size_Mat];
         dist.f[dirSE  ] = &distributions[dirNW  *size_Mat];
         dist.f[dirBW  ] = &distributions[dirTE  *size_Mat];
         dist.f[dirTE  ] = &distributions[dirBW  *size_Mat];
         dist.f[dirTW  ] = &distributions[dirBE  *size_Mat];
         dist.f[dirBE  ] = &distributions[dirTW  *size_Mat];
         dist.f[dirBS  ] = &distributions[dirTN  *size_Mat];
         dist.f[dirTN  ] = &distributions[dirBS  *size_Mat];
         dist.f[dirTS  ] = &distributions[dirBN  *size_Mat];
         dist.f[dirBN  ] = &distributions[dirTS  *size_Mat];
         dist.f[dirREST] = &distributions[dirREST*size_Mat];
         dist.f[dirTNE ] = &distributions[dirBSW *size_Mat];
         dist.f[dirTSW ] = &distributions[dirBNE *size_Mat];
         dist.f[dirTSE ] = &distributions[dirBNW *size_Mat];
         dist.f[dirTNW ] = &distributions[dirBSE *size_Mat];
         dist.f[dirBNE ] = &distributions[dirTSW *size_Mat];
         dist.f[dirBSW ] = &distributions[dirTNE *size_Mat];
         dist.f[dirBSE ] = &distributions[dirTNW *size_Mat];
         dist.f[dirBNW ] = &distributions[dirTSE *size_Mat];
      }
	  ////////////////////////////////////////////////////////////////////////////////
	  //! - rewrite distributions if there is a sub-grid distance (q) in same direction
	  real q;
      q = q_dirE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirW  ])[kw  ]=f_E   + c4o9  * (-VeloX);	
      q = q_dirW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirE  ])[ke  ]=f_W   + c4o9  * ( VeloX);	
      q = q_dirN[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirS  ])[ks  ]=f_N   + c4o9  * (-VeloY);	
      q = q_dirS[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirN  ])[kn  ]=f_S   + c4o9  * ( VeloY);	
      q = q_dirT[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirB  ])[kb  ]=f_T   + c4o9  * (-VeloZ);
      q = q_dirB[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirT  ])[kt  ]=f_B   + c4o9  * ( VeloZ);
      q = q_dirNE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirSW ])[ksw ]=f_NE  + c1o9  * (-VeloX - VeloY);
	  q = q_dirSW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirNE ])[kne ]=f_SW  + c1o9  * ( VeloX + VeloY);
	  q = q_dirSE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirNW ])[knw ]=f_SE  + c1o9  * (-VeloX + VeloY);
	  q = q_dirNW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirSE ])[kse ]=f_NW  + c1o9  * ( VeloX - VeloY);
	  q = q_dirTE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirBW ])[kbw ]=f_TE  + c1o9  * (-VeloX - VeloZ);
	  q = q_dirBW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirTE ])[kte ]=f_BW  + c1o9  * ( VeloX + VeloZ);
	  q = q_dirBE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirTW ])[ktw ]=f_BE  + c1o9  * (-VeloX + VeloZ);
	  q = q_dirTW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirBE ])[kbe ]=f_TW  + c1o9  * ( VeloX - VeloZ);
	  q = q_dirTN[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirBS ])[kbs ]=f_TN  + c1o9  * (-VeloY - VeloZ);
	  q = q_dirBS[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirTN ])[ktn ]=f_BS  + c1o9  * ( VeloY + VeloZ);
	  q = q_dirBN[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirTS ])[kts ]=f_BN  + c1o9  * (-VeloY + VeloZ);
	  q = q_dirTS[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirBN ])[kbn ]=f_TS  + c1o9  * ( VeloY - VeloZ);
      q = q_dirTNE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirBSW])[kbsw]=f_TNE + c1o36 * (-VeloX - VeloY - VeloZ);
      q = q_dirBSW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirTNE])[ktne]=f_BSW + c1o36 * ( VeloX + VeloY + VeloZ);
      q = q_dirBNE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirTSW])[ktsw]=f_BNE + c1o36 * (-VeloX - VeloY + VeloZ);
      q = q_dirTSW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirBNE])[kbne]=f_TSW + c1o36 * ( VeloX + VeloY - VeloZ);
      q = q_dirTSE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirBNW])[kbnw]=f_TSE + c1o36 * (-VeloX + VeloY - VeloZ);
      q = q_dirBNW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirTSE])[ktse]=f_BNW + c1o36 * ( VeloX - VeloY + VeloZ);
      q = q_dirBSE[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirTNW])[ktnw]=f_BSE + c1o36 * (-VeloX + VeloY + VeloZ);
      q = q_dirTNW[k];	if (q>=c0o1 && q<=c1o1)	(dist.f[dirBSE])[kbse]=f_TNW + c1o36 * ( VeloX - VeloY - VeloZ);
   }
}



