#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;

////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBInit27( int myid,
                                     int numprocs,
                                     real u0,
                                     unsigned int* geoD,
                                     unsigned int* neighborX,
                                     unsigned int* neighborY,
                                     unsigned int* neighborZ,
                                     real* vParabel,
                                     unsigned int size_Mat,
                                     unsigned int grid_nx, 
                                     unsigned int grid_ny, 
                                     unsigned int grid_nz, 
                                     real* DD,
                                     int lev,
                                     int maxlev)
{
   Distributions27 D;
   D.f[E   ] = &DD[E   *size_Mat];
   D.f[W   ] = &DD[W   *size_Mat];
   D.f[N   ] = &DD[N   *size_Mat];
   D.f[S   ] = &DD[S   *size_Mat];
   D.f[T   ] = &DD[T   *size_Mat];
   D.f[B   ] = &DD[B   *size_Mat];
   D.f[NE  ] = &DD[NE  *size_Mat];
   D.f[SW  ] = &DD[SW  *size_Mat];
   D.f[SE  ] = &DD[SE  *size_Mat];
   D.f[NW  ] = &DD[NW  *size_Mat];
   D.f[TE  ] = &DD[TE  *size_Mat];
   D.f[BW  ] = &DD[BW  *size_Mat];
   D.f[BE  ] = &DD[BE  *size_Mat];
   D.f[TW  ] = &DD[TW  *size_Mat];
   D.f[TN  ] = &DD[TN  *size_Mat];
   D.f[BS  ] = &DD[BS  *size_Mat];
   D.f[BN  ] = &DD[BN  *size_Mat];
   D.f[TS  ] = &DD[TS  *size_Mat];
   D.f[REST] = &DD[REST*size_Mat];
   D.f[TNE ] = &DD[TNE *size_Mat];
   D.f[TSW ] = &DD[TSW *size_Mat];
   D.f[TSE ] = &DD[TSE *size_Mat];
   D.f[TNW ] = &DD[TNW *size_Mat];
   D.f[BNE ] = &DD[BNE *size_Mat];
   D.f[BSW ] = &DD[BSW *size_Mat];
   D.f[BSE ] = &DD[BSE *size_Mat];
   D.f[BNW ] = &DD[BNW *size_Mat];
   ////////////////////////////////////////////////////////////////////////////////
   unsigned int  k;                   // Zugriff auf arrays im device
   //
   unsigned int tx = threadIdx.x;     // Thread index = lokaler i index
   unsigned int by = blockIdx.x;      // Block index x
   unsigned int bz = blockIdx.y;      // Block index y
   unsigned int  x = tx + STARTOFFX;  // Globaler x-Index 
   unsigned int  y = by + STARTOFFY;  // Globaler y-Index 
   unsigned int  z = bz + STARTOFFZ;  // Globaler z-Index 

   const unsigned sizeX = blockDim.x;
   const unsigned sizeY = gridDim.x;
   const unsigned nx = sizeX + 2 * STARTOFFX;
   const unsigned ny = sizeY + 2 * STARTOFFY;

   k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////
   geoD[k] = GEO_FLUID;
   if (lev==0)
   {
      if( by == 0 || by == grid_ny-1 || tx == 0 || tx == grid_nx-1 )             
         geoD[k] = GEO_SOLID;
      else if( bz == grid_nz-1 && myid == numprocs - 1 && geoD[k] != GEO_SOLID )
         geoD[k] = GEO_PRESS;				 
      else if( bz == 0 && myid == 0 && geoD[k] != GEO_SOLID)
         geoD[k] = GEO_SOLID;//GEO_VELO;
   }
   else if (lev==maxlev-1)
   {
      unsigned int centerX = grid_nx / 2;
      unsigned int centerY = grid_ny / 2;
      unsigned int centerZ = grid_nz / 2;
      real        radius  = grid_ny / 2.56;

      unsigned int distSq = (centerX-tx)*(centerX-tx)+(centerY-by)*(centerY-by)+(centerZ-bz)*(centerZ-bz);
      real radiSq = radius*radius;

      if( distSq < radiSq)        geoD[k] = GEO_SOLID;
   }
   //////////////////////////////////////////////////////////////////////////
   real drho = c0o1;
   real  vx1 = c0o1;
   real  vx2 = c0o1;
   real  vx3 = u0;
   vParabel[k] = vx3;
   ////////////////////////////////////////////////////////////////////////////////
   //index
   unsigned int nxny = nx*ny;
   ////////////////////////////////////////////////////////////////////////////////
   //neighborX[k]      = k+1;
   //neighborY[k+1]    = k+nx+1;
   //neighborZ[k+1]    = k+nxny+1;
   //neighborY[k]      = k+nx;
   //neighborX[k+nx]   = k+nx+1;
   //neighborZ[k+nx]   = k+nx+nxny;
   //neighborZ[k]      = k+nxny;
   //neighborX[k+nxny] = k+nxny+1;
   //neighborY[k+nxny] = k+nxny+nx;
   ////////////////////////////////////////////////////////////////////////////////
   unsigned int kzero= k;
   unsigned int ke   = k;
   unsigned int kw   = k + 1;
   unsigned int kn   = k;
   unsigned int ks   = k + nx;
   unsigned int kt   = k;
   unsigned int kb   = k + nxny;
   unsigned int ksw  = k + nx + 1;
   unsigned int kne  = k;
   unsigned int kse  = k + nx;
   unsigned int knw  = k + 1;
   unsigned int kbw  = k + nxny + 1;
   unsigned int kte  = k;
   unsigned int kbe  = k + nxny;
   unsigned int ktw  = k + 1;
   unsigned int kbs  = k + nxny + nx;
   unsigned int ktn  = k;
   unsigned int kbn  = k + nxny;
   unsigned int kts  = k + nx;
   unsigned int ktse = k + nx;
   unsigned int kbnw = k + nxny + 1;
   unsigned int ktnw = k + 1;
   unsigned int kbse = k + nxny + nx;
   unsigned int ktsw = k + nx + 1;
   unsigned int kbne = k + nxny;
   unsigned int ktne = k;
   unsigned int kbsw = k + nxny + nx + 1;
   //////////////////////////////////////////////////////////////////////////

   real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

   (D.f[REST])[kzero] =   c8o27* (drho-cu_sq);
   (D.f[E   ])[ke   ] =   c2o27* (drho+c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq);
   (D.f[W   ])[kw   ] =   c2o27* (drho+c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq);
   (D.f[N   ])[kn   ] =   c2o27* (drho+c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq);
   (D.f[S   ])[ks   ] =   c2o27* (drho+c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq);
   (D.f[T   ])[kt   ] =   c2o27* (drho+c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq);
   (D.f[B   ])[kb   ] =   c2o27* (drho+c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq);
   (D.f[NE  ])[kne  ] =   c1o54* (drho+c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq);
   (D.f[SW  ])[ksw  ] =   c1o54* (drho+c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq);
   (D.f[SE  ])[kse  ] =   c1o54* (drho+c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq);
   (D.f[NW  ])[knw  ] =   c1o54* (drho+c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq);
   (D.f[TE  ])[kte  ] =   c1o54* (drho+c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq);
   (D.f[BW  ])[kbw  ] =   c1o54* (drho+c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq);
   (D.f[BE  ])[kbe  ] =   c1o54* (drho+c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq);
   (D.f[TW  ])[ktw  ] =   c1o54* (drho+c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq);
   (D.f[TN  ])[ktn  ] =   c1o54* (drho+c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq);
   (D.f[BS  ])[kbs  ] =   c1o54* (drho+c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq);
   (D.f[BN  ])[kbn  ] =   c1o54* (drho+c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq);
   (D.f[TS  ])[kts  ] =   c1o54* (drho+c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq);
   (D.f[TNE ])[ktne ] =   c1o216*(drho+c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq);
   (D.f[BSW ])[kbsw ] =   c1o216*(drho+c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq);
   (D.f[BNE ])[kbne ] =   c1o216*(drho+c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq);
   (D.f[TSW ])[ktsw ] =   c1o216*(drho+c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq);
   (D.f[TSE ])[ktse ] =   c1o216*(drho+c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq);
   (D.f[BNW ])[kbnw ] =   c1o216*(drho+c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq);
   (D.f[BSE ])[kbse ] =   c1o216*(drho+c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq);
   (D.f[TNW ])[ktnw ] =   c1o216*(drho+c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq);

}
////////////////////////////////////////////////////////////////////////////////










////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBInitNonEqPartSP27( unsigned int* neighborX,
                                                unsigned int* neighborY,
                                                unsigned int* neighborZ,
                                                unsigned int* neighborWSB,
                                                unsigned int* geoD,
                                                real* rho,
                                                real* ux,
                                                real* uy,
                                                real* uz,
                                                unsigned int size_Mat,
                                                real* DD,
                                                real omega,
                                                bool EvenOrOdd)
{
    ////////////////////////////////////////////////////////////////////////////////
    const unsigned  x = threadIdx.x;  // Globaler x-Index 
    const unsigned  y = blockIdx.x;   // Globaler y-Index 
    const unsigned  z = blockIdx.y;   // Globaler z-Index 
    
    const unsigned nx = blockDim.x;
    const unsigned ny = gridDim.x;
    
    const unsigned k = nx*(ny*z + y) + x;
    //////////////////////////////////////////////////////////////////////////
    
    if(k<size_Mat)
    {
        ////////////////////////////////////////////////////////////////////////////////
        unsigned int BC;
        BC = geoD[k];

        if( BC != GEO_SOLID &&  BC != GEO_VOID)
        {
            Distributions27 D;
            if (EvenOrOdd==true)
            {
                D.f[E   ] = &DD[E   *size_Mat];
                D.f[W   ] = &DD[W   *size_Mat];
                D.f[N   ] = &DD[N   *size_Mat];
                D.f[S   ] = &DD[S   *size_Mat];
                D.f[T   ] = &DD[T   *size_Mat];
                D.f[B   ] = &DD[B   *size_Mat];
                D.f[NE  ] = &DD[NE  *size_Mat];
                D.f[SW  ] = &DD[SW  *size_Mat];
                D.f[SE  ] = &DD[SE  *size_Mat];
                D.f[NW  ] = &DD[NW  *size_Mat];
                D.f[TE  ] = &DD[TE  *size_Mat];
                D.f[BW  ] = &DD[BW  *size_Mat];
                D.f[BE  ] = &DD[BE  *size_Mat];
                D.f[TW  ] = &DD[TW  *size_Mat];
                D.f[TN  ] = &DD[TN  *size_Mat];
                D.f[BS  ] = &DD[BS  *size_Mat];
                D.f[BN  ] = &DD[BN  *size_Mat];
                D.f[TS  ] = &DD[TS  *size_Mat];
                D.f[REST] = &DD[REST*size_Mat];
                D.f[TNE ] = &DD[TNE *size_Mat];
                D.f[TSW ] = &DD[TSW *size_Mat];
                D.f[TSE ] = &DD[TSE *size_Mat];
                D.f[TNW ] = &DD[TNW *size_Mat];
                D.f[BNE ] = &DD[BNE *size_Mat];
                D.f[BSW ] = &DD[BSW *size_Mat];
                D.f[BSE ] = &DD[BSE *size_Mat];
                D.f[BNW ] = &DD[BNW *size_Mat];
            }
            else
            {
                D.f[W   ] = &DD[E   *size_Mat];
                D.f[E   ] = &DD[W   *size_Mat];
                D.f[S   ] = &DD[N   *size_Mat];
                D.f[N   ] = &DD[S   *size_Mat];
                D.f[B   ] = &DD[T   *size_Mat];
                D.f[T   ] = &DD[B   *size_Mat];
                D.f[SW  ] = &DD[NE  *size_Mat];
                D.f[NE  ] = &DD[SW  *size_Mat];
                D.f[NW  ] = &DD[SE  *size_Mat];
                D.f[SE  ] = &DD[NW  *size_Mat];
                D.f[BW  ] = &DD[TE  *size_Mat];
                D.f[TE  ] = &DD[BW  *size_Mat];
                D.f[TW  ] = &DD[BE  *size_Mat];
                D.f[BE  ] = &DD[TW  *size_Mat];
                D.f[BS  ] = &DD[TN  *size_Mat];
                D.f[TN  ] = &DD[BS  *size_Mat];
                D.f[TS  ] = &DD[BN  *size_Mat];
                D.f[BN  ] = &DD[TS  *size_Mat];
                D.f[REST] = &DD[REST*size_Mat];
                D.f[BSW ] = &DD[TNE *size_Mat];
                D.f[BNE ] = &DD[TSW *size_Mat];
                D.f[BNW ] = &DD[TSE *size_Mat];
                D.f[BSE ] = &DD[TNW *size_Mat];
                D.f[TSW ] = &DD[BNE *size_Mat];
                D.f[TNE ] = &DD[BSW *size_Mat];
                D.f[TNW ] = &DD[BSE *size_Mat];
                D.f[TSE ] = &DD[BNW *size_Mat];
            }
            //////////////////////////////////////////////////////////////////////////
            real drho = rho[k];//0.0f;//
            real  vx1 = ux[k]; //0.0f;//
            real  vx2 = uy[k]; //0.0f;//
            real  vx3 = uz[k]; //0.0f;//
            //////////////////////////////////////////////////////////////////////////
            //index
            //////////////////////////////////////////////////////////////////////////
            unsigned int kzero= k;
            unsigned int ke   = k;
            unsigned int kw   = neighborX[k];
            unsigned int kn   = k;
            unsigned int ks   = neighborY[k];
            unsigned int kt   = k;
            unsigned int kb   = neighborZ[k];
            unsigned int ksw  = neighborY[kw];
            unsigned int kne  = k;
            unsigned int kse  = ks;
            unsigned int knw  = kw;
            unsigned int kbw  = neighborZ[kw];
            unsigned int kte  = k;
            unsigned int kbe  = kb;
            unsigned int ktw  = kw;
            unsigned int kbs  = neighborZ[ks];
            unsigned int ktn  = k;
            unsigned int kbn  = kb;
            unsigned int kts  = ks;
            unsigned int ktse = ks;
            unsigned int kbnw = kbw;
            unsigned int ktnw = kw;
            unsigned int kbse = kbs;
            unsigned int ktsw = ksw;
            unsigned int kbne = kb;
            unsigned int ktne = k;
            unsigned int kbsw = neighborZ[ksw];
	        //////////////////////////////////////////////////////////////////////////////
	        //neighbor index
	        uint kPx   = neighborX[k];
	        uint kPy   = neighborY[k];
	        uint kPz   = neighborZ[k];
	        uint kMxyz = neighborWSB[k];
	        uint kMx   = neighborZ[neighborY[kMxyz]];
	        uint kMy   = neighborZ[neighborX[kMxyz]];
	        uint kMz   = neighborY[neighborX[kMxyz]];
            //////////////////////////////////////////////////////////////////////////
	        //getVeloX//
	        real vx1NeighborPx = ux[kPx];
	        real vx1NeighborMx = ux[kMx];
	        real vx1NeighborPy = ux[kPy];
	        real vx1NeighborMy = ux[kMy];
	        real vx1NeighborPz = ux[kPz];
	        real vx1NeighborMz = ux[kMz];
	        //getVeloY//
	        real vx2NeighborPx = uy[kPx];
	        real vx2NeighborMx = uy[kMx];
	        real vx2NeighborPy = uy[kPy];
	        real vx2NeighborMy = uy[kMy];
	        real vx2NeighborPz = uy[kPz];
	        real vx2NeighborMz = uy[kMz];
	        //getVeloZ//
	        real vx3NeighborPx = uz[kPx];
	        real vx3NeighborMx = uz[kMx];
	        real vx3NeighborPy = uz[kPy];
	        real vx3NeighborMy = uz[kMy];
	        real vx3NeighborPz = uz[kPz];
	        real vx3NeighborMz = uz[kMz];
            //////////////////////////////////////////////////////////////////////////

	        real dvx1dx = (vx1NeighborPx - vx1NeighborMx) / c2o1;
	        real dvx1dy = (vx1NeighborPy - vx1NeighborMy) / c2o1;
	        real dvx1dz = (vx1NeighborPz - vx1NeighborMz) / c2o1;

	        real dvx2dx = (vx2NeighborPx - vx2NeighborMx) / c2o1;
	        real dvx2dy = (vx2NeighborPy - vx2NeighborMy) / c2o1;
	        real dvx2dz = (vx2NeighborPz - vx2NeighborMz) / c2o1;

	        real dvx3dx = (vx3NeighborPx - vx3NeighborMx) / c2o1;
	        real dvx3dy = (vx3NeighborPy - vx3NeighborMy) / c2o1;
	        real dvx3dz = (vx3NeighborPz - vx3NeighborMz) / c2o1;

            //////////////////////////////////////////////////////////////////////////

            // the following code is copy and pasted from VirtualFluidsCore/Visitors/InitDistributionsBlockVisitor.cpp
            // i.e. Konstantins code

            real ax = dvx1dx;
            real ay = dvx1dy;
            real az = dvx1dz;

            real bx = dvx2dx;
            real by = dvx2dy;
            real bz = dvx2dz;

            real cx = dvx3dx;
            real cy = dvx3dy;
            real cz = dvx3dz;

            real eps_new = c1o1;
            real op      = c1o1;
            real o       = omega;

            real f_E    = eps_new *((5.*ax*omega + 5.*by*o + 5.*cz*o - 8.*ax*op + 4.*by*op + 4.*cz*op)/(54.*o*op));

            real f_N    =    f_E   + eps_new *((2.*(ax - by))/(9.*o));
            real f_T    =    f_E   + eps_new *((2.*(ax - cz))/(9.*o));
            real f_NE   =            eps_new *(-(5.*cz*o + 3.*(ay + bx)*op - 2.*cz*op + ax*(5.*o + op) + by*(5.*o + op))/(54.*o*op));
            real f_SE   =    f_NE  + eps_new *((  ay + bx )/(9.*o));
            real f_TE   =            eps_new *(-(5.*cz*o + by*(5.*o - 2.*op) + 3.*(az + cx)*op + cz*op + ax*(5.*o + op))/(54.*o*op));
            real f_BE   =    f_TE  + eps_new *((  az + cx )/(9.*o));
            real f_TN   =            eps_new *(-(5.*ax*o + 5.*by*o + 5.*cz*o - 2.*ax*op + by*op + 3.*bz*op + 3.*cy*op + cz*op)/(54.*o*op));
            real f_BN   =    f_TN  + eps_new *((  bz + cy )/(9.*o));
            real f_ZERO =            eps_new *((5.*(ax + by + cz))/(9.*op));
            real f_TNE  =            eps_new *(-(ay + az + bx + bz + cx + cy)/(72.*o));
            real f_TSW  =  - f_TNE - eps_new *((ay + bx)/(36.*o));
            real f_TSE  =  - f_TNE - eps_new *((az + cx)/(36.*o));
            real f_TNW  =  - f_TNE - eps_new *((bz + cy)/(36.*o));

            //////////////////////////////////////////////////////////////////////////
            real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);
            
            (D.f[REST])[kzero] =   c8o27* (drho-cu_sq);
            (D.f[E   ])[ke   ] =   c2o27* (drho+c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq);
            (D.f[W   ])[kw   ] =   c2o27* (drho+c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq);
            (D.f[N   ])[kn   ] =   c2o27* (drho+c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq);
            (D.f[S   ])[ks   ] =   c2o27* (drho+c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq);
            (D.f[T   ])[kt   ] =   c2o27* (drho+c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq);
            (D.f[B   ])[kb   ] =   c2o27* (drho+c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq);
            (D.f[NE  ])[kne  ] =   c1o54* (drho+c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq);
            (D.f[SW  ])[ksw  ] =   c1o54* (drho+c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq);
            (D.f[SE  ])[kse  ] =   c1o54* (drho+c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq);
            (D.f[NW  ])[knw  ] =   c1o54* (drho+c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq);
            (D.f[TE  ])[kte  ] =   c1o54* (drho+c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq);
            (D.f[BW  ])[kbw  ] =   c1o54* (drho+c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq);
            (D.f[BE  ])[kbe  ] =   c1o54* (drho+c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq);
            (D.f[TW  ])[ktw  ] =   c1o54* (drho+c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq);
            (D.f[TN  ])[ktn  ] =   c1o54* (drho+c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq);
            (D.f[BS  ])[kbs  ] =   c1o54* (drho+c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq);
            (D.f[BN  ])[kbn  ] =   c1o54* (drho+c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq);
            (D.f[TS  ])[kts  ] =   c1o54* (drho+c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq);
            (D.f[TNE ])[ktne ] =   c1o216*(drho+c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq);
            (D.f[BSW ])[kbsw ] =   c1o216*(drho+c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq);
            (D.f[BNE ])[kbne ] =   c1o216*(drho+c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq);
            (D.f[TSW ])[ktsw ] =   c1o216*(drho+c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq);
            (D.f[TSE ])[ktse ] =   c1o216*(drho+c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq);
            (D.f[BNW ])[kbnw ] =   c1o216*(drho+c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq);
            (D.f[BSE ])[kbse ] =   c1o216*(drho+c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq);
            (D.f[TNW ])[ktnw ] =   c1o216*(drho+c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq);

            //////////////////////////////////////////////////////////////////////////

            (D.f[REST])[kzero] += f_ZERO;
            (D.f[E   ])[ke   ] += f_E   ;
            (D.f[W   ])[kw   ] += f_E   ;
            (D.f[N   ])[kn   ] += f_N   ;
            (D.f[S   ])[ks   ] += f_N   ;
            (D.f[T   ])[kt   ] += f_T   ;
            (D.f[B   ])[kb   ] += f_T   ;
            (D.f[NE  ])[kne  ] += f_NE  ;
            (D.f[SW  ])[ksw  ] += f_NE  ;
            (D.f[SE  ])[kse  ] += f_SE  ;
            (D.f[NW  ])[knw  ] += f_SE  ;
            (D.f[TE  ])[kte  ] += f_TE  ;
            (D.f[BW  ])[kbw  ] += f_TE  ;
            (D.f[BE  ])[kbe  ] += f_BE  ;
            (D.f[TW  ])[ktw  ] += f_BE  ;
            (D.f[TN  ])[ktn  ] += f_TN  ;
            (D.f[BS  ])[kbs  ] += f_TN  ;
            (D.f[BN  ])[kbn  ] += f_BN  ;
            (D.f[TS  ])[kts  ] += f_BN  ;
            (D.f[TNE ])[ktne ] += f_TNE ;
            (D.f[BSW ])[kbsw ] += f_TNE ;
            (D.f[BNE ])[kbne ] += f_TSW ;
            (D.f[TSW ])[ktsw ] += f_TSW ;
            (D.f[TSE ])[ktse ] += f_TSE ;
            (D.f[BNW ])[kbnw ] += f_TSE ;
            (D.f[BSE ])[kbse ] += f_TNW ;
            (D.f[TNW ])[ktnw ] += f_TNW ;

            //////////////////////////////////////////////////////////////////////////
        }
	    else
	    {
		    //////////////////////////////////////////////////////////////////////////
		    Distributions27 D;
		    D.f[REST] = &DD[REST*size_Mat];
		    //////////////////////////////////////////////////////////////////////////
		    (D.f[REST])[k] = c96o1;
		    //////////////////////////////////////////////////////////////////////////
	    }
   }
}











































