#include "hip/hip_runtime.h"
//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ / 
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include "Core/RealConstants.h"
#include "math.h"


////////////////////////////////////////////////////////////////////////////////
inline __device__ void forwardInverseChimeraWithK(real &mfa, real &mfb, real &mfc, real vv, real v2, real Kinverse, real K) {
	real m2 = mfa + mfc;
	real m1 = mfc - mfa;
	real m0 = m2 + mfb;
	mfa = m0;
	m0 *= Kinverse;
	m0 += c1o1;
	mfb = (m1*Kinverse - m0 * vv) * K;
	mfc = ((m2 - c2o1*	m1 * vv)*Kinverse + v2 * m0) * K;
}

inline __device__ void backwardInverseChimeraWithK(real &mfa, real &mfb, real &mfc, real vv, real v2, real Kinverse, real K) {
	real m0 = (((mfc - mfb) * c1o2 + mfb *  vv)*Kinverse + (mfa*Kinverse + c1o1) * (v2 - vv) * c1o2) * K;
	real m1 = (((mfa - mfc) -  c2o1 * mfb *  vv)*Kinverse + (mfa*Kinverse + c1o1) * (           -v2)) * K;
	mfc     = (((mfc + mfb) * c1o2 + mfb *  vv)*Kinverse + (mfa*Kinverse + c1o1) * (v2 + vv) * c1o2) * K;
	mfa = m0;
	mfb = m1;
}
////////////////////////////////////////////////////////////////////////////////





inline __device__ void forwardChimeraWithK(real &mfa, real &mfb, real &mfc, real vv, real v2, real K) {

	real m2 = mfa + mfc;
	real m1 = mfc - mfa;
	real m0 = m2 + mfb;
	mfa = m0;
	//m0     += K;
	mfb = (m1 - K*vv) - m0 * vv;
	mfc = ((m2 - c2o1*	m1 * vv) + v2*K) + v2 * m0;
	//m0 += K;
	//mfb = m1 - m0 * vv;
	//mfc = m2 - two*	m1 * vv + v2 * m0;
}

inline __device__ void forwardChimera(real &mfa, real &mfb, real &mfc, real vv, real v2) {
	real m1 = (mfa + mfc) + mfb;
	real m2 = mfc - mfa;
	mfc = (mfc + mfa) + (v2*m1 - c2o1*vv*m2);
	mfb = m2 - vv*m1;
	mfa = m1;
}


inline __device__ void backwardChimera(real &mfa, real &mfb, real &mfc, real vv, real v2) {
	real ma = (mfc + mfa*(v2 - vv))*c1o2 + mfb*(vv - c1o2);
	real mb = ((mfa - mfc) - mfa*v2) - c2o1*mfb*vv;
	mfc = (mfc + mfa*(v2 + vv))*c1o2 + mfb*(vv + c1o2);
	mfb = mb;
	mfa = ma;
}


inline __device__ void backwardChimeraWithK(real &mfa, real &mfb, real &mfc, real vv, real v2, real K) {
	real  m0 = (mfc - mfb)* c1o2 + mfb * (vv)+(mfa + K) * (v2 - vv) * c1o2;
	real m1 = (mfa - mfc) - c2o1* mfb * vv + (mfa + K) * (-v2);
	mfc = (mfc + mfb)* c1o2 + mfb * (vv)+(mfa + K) * (v2 + vv) * c1o2;
	mfa = m0;
	mfb = m1;

}






////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void Cumulant_One_preconditioned_errorDiffusion_chim_Comp_SP_27(
	real omega,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* DDStart,
	int size_Mat,
	int level,
	real* forces,
	bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if (BC >= GEO_FLUID/*(BC != GEO_SOLID) && (BC != GEO_VOID)*/)
		{
			Distributions27 D;
			if (EvenOrOdd == true)
			{
				D.f[dirE] = &DDStart[dirE   *size_Mat];
				D.f[dirW] = &DDStart[dirW   *size_Mat];
				D.f[dirN] = &DDStart[dirN   *size_Mat];
				D.f[dirS] = &DDStart[dirS   *size_Mat];
				D.f[dirT] = &DDStart[dirT   *size_Mat];
				D.f[dirB] = &DDStart[dirB   *size_Mat];
				D.f[dirNE] = &DDStart[dirNE  *size_Mat];
				D.f[dirSW] = &DDStart[dirSW  *size_Mat];
				D.f[dirSE] = &DDStart[dirSE  *size_Mat];
				D.f[dirNW] = &DDStart[dirNW  *size_Mat];
				D.f[dirTE] = &DDStart[dirTE  *size_Mat];
				D.f[dirBW] = &DDStart[dirBW  *size_Mat];
				D.f[dirBE] = &DDStart[dirBE  *size_Mat];
				D.f[dirTW] = &DDStart[dirTW  *size_Mat];
				D.f[dirTN] = &DDStart[dirTN  *size_Mat];
				D.f[dirBS] = &DDStart[dirBS  *size_Mat];
				D.f[dirBN] = &DDStart[dirBN  *size_Mat];
				D.f[dirTS] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirTNE] = &DDStart[dirTNE *size_Mat];
				D.f[dirTSW] = &DDStart[dirTSW *size_Mat];
				D.f[dirTSE] = &DDStart[dirTSE *size_Mat];
				D.f[dirTNW] = &DDStart[dirTNW *size_Mat];
				D.f[dirBNE] = &DDStart[dirBNE *size_Mat];
				D.f[dirBSW] = &DDStart[dirBSW *size_Mat];
				D.f[dirBSE] = &DDStart[dirBSE *size_Mat];
				D.f[dirBNW] = &DDStart[dirBNW *size_Mat];
			}
			else
			{
				D.f[dirW] = &DDStart[dirE   *size_Mat];
				D.f[dirE] = &DDStart[dirW   *size_Mat];
				D.f[dirS] = &DDStart[dirN   *size_Mat];
				D.f[dirN] = &DDStart[dirS   *size_Mat];
				D.f[dirB] = &DDStart[dirT   *size_Mat];
				D.f[dirT] = &DDStart[dirB   *size_Mat];
				D.f[dirSW] = &DDStart[dirNE  *size_Mat];
				D.f[dirNE] = &DDStart[dirSW  *size_Mat];
				D.f[dirNW] = &DDStart[dirSE  *size_Mat];
				D.f[dirSE] = &DDStart[dirNW  *size_Mat];
				D.f[dirBW] = &DDStart[dirTE  *size_Mat];
				D.f[dirTE] = &DDStart[dirBW  *size_Mat];
				D.f[dirTW] = &DDStart[dirBE  *size_Mat];
				D.f[dirBE] = &DDStart[dirTW  *size_Mat];
				D.f[dirBS] = &DDStart[dirTN  *size_Mat];
				D.f[dirTN] = &DDStart[dirBS  *size_Mat];
				D.f[dirTS] = &DDStart[dirBN  *size_Mat];
				D.f[dirBN] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirBSW] = &DDStart[dirTNE *size_Mat];
				D.f[dirBNE] = &DDStart[dirTSW *size_Mat];
				D.f[dirBNW] = &DDStart[dirTSE *size_Mat];
				D.f[dirBSE] = &DDStart[dirTNW *size_Mat];
				D.f[dirTSW] = &DDStart[dirBNE *size_Mat];
				D.f[dirTNE] = &DDStart[dirBSW *size_Mat];
				D.f[dirTNW] = &DDStart[dirBSE *size_Mat];
				D.f[dirTSE] = &DDStart[dirBNW *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			//unsigned int kzero= k;
			//unsigned int ke   = k;
			unsigned int kw = neighborX[k];
			//unsigned int kn   = k;
			unsigned int ks = neighborY[k];
			//unsigned int kt   = k;
			unsigned int kb = neighborZ[k];
			unsigned int ksw = neighborY[kw];
			//unsigned int kne  = k;
			//unsigned int kse  = ks;
			//unsigned int knw  = kw;
			unsigned int kbw = neighborZ[kw];
			//unsigned int kte  = k;
			//unsigned int kbe  = kb;
			//unsigned int ktw  = kw;
			unsigned int kbs = neighborZ[ks];
			//unsigned int ktn  = k;
			//unsigned int kbn  = kb;
			//unsigned int kts  = ks;
			//unsigned int ktse = ks;
			//unsigned int kbnw = kbw;
			//unsigned int ktnw = kw;
			//unsigned int kbse = kbs;
			//unsigned int ktsw = ksw;
			//unsigned int kbne = kb;
			//unsigned int ktne = k;
			unsigned int kbsw = neighborZ[ksw];



			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[dirE])[k];//[ke   ];// +  c2over27 ;(D.f[dirE   ])[k  ];//ke
			real mfabb = (D.f[dirW])[kw];//[kw   ];// +  c2over27 ;(D.f[dirW   ])[kw ];
			real mfbcb = (D.f[dirN])[k];//[kn   ];// +  c2over27 ;(D.f[dirN   ])[k  ];//kn
			real mfbab = (D.f[dirS])[ks];//[ks   ];// +  c2over27 ;(D.f[dirS   ])[ks ];
			real mfbbc = (D.f[dirT])[k];//[kt   ];// +  c2over27 ;(D.f[dirT   ])[k  ];//kt
			real mfbba = (D.f[dirB])[kb];//[kb   ];// +  c2over27 ;(D.f[dirB   ])[kb ];
			real mfccb = (D.f[dirNE])[k];//[kne  ];// +  c1over54 ;(D.f[dirNE  ])[k  ];//kne
			real mfaab = (D.f[dirSW])[ksw];//[ksw  ];// +  c1over54 ;(D.f[dirSW  ])[ksw];
			real mfcab = (D.f[dirSE])[ks];//[kse  ];// +  c1over54 ;(D.f[dirSE  ])[ks ];//kse
			real mfacb = (D.f[dirNW])[kw];//[knw  ];// +  c1over54 ;(D.f[dirNW  ])[kw ];//knw
			real mfcbc = (D.f[dirTE])[k];//[kte  ];// +  c1over54 ;(D.f[dirTE  ])[k  ];//kte
			real mfaba = (D.f[dirBW])[kbw];//[kbw  ];// +  c1over54 ;(D.f[dirBW  ])[kbw];
			real mfcba = (D.f[dirBE])[kb];//[kbe  ];// +  c1over54 ;(D.f[dirBE  ])[kb ];//kbe
			real mfabc = (D.f[dirTW])[kw];//[ktw  ];// +  c1over54 ;(D.f[dirTW  ])[kw ];//ktw
			real mfbcc = (D.f[dirTN])[k];//[ktn  ];// +  c1over54 ;(D.f[dirTN  ])[k  ];//ktn
			real mfbaa = (D.f[dirBS])[kbs];//[kbs  ];// +  c1over54 ;(D.f[dirBS  ])[kbs];
			real mfbca = (D.f[dirBN])[kb];//[kbn  ];// +  c1over54 ;(D.f[dirBN  ])[kb ];//kbn
			real mfbac = (D.f[dirTS])[ks];//[kts  ];// +  c1over54 ;(D.f[dirTS  ])[ks ];//kts
			real mfbbb = (D.f[dirZERO])[k];//[kzero];// +  c8over27 ;(D.f[dirZERO])[k  ];//kzero
			real mfccc = (D.f[dirTNE])[k];//[ktne ];// +  c1over216;(D.f[dirTNE ])[k  ];//ktne
			real mfaac = (D.f[dirTSW])[ksw];//[ktsw ];// +  c1over216;(D.f[dirTSW ])[ksw];//ktsw
			real mfcac = (D.f[dirTSE])[ks];//[ktse ];// +  c1over216;(D.f[dirTSE ])[ks ];//ktse
			real mfacc = (D.f[dirTNW])[kw];//[ktnw ];// +  c1over216;(D.f[dirTNW ])[kw ];//ktnw
			real mfcca = (D.f[dirBNE])[kb];//[kbne ];// +  c1over216;(D.f[dirBNE ])[kb ];//kbne
			real mfaaa = (D.f[dirBSW])[kbsw];//[kbsw ];// +  c1over216;(D.f[dirBSW ])[kbsw];
			real mfcaa = (D.f[dirBSE])[kbs];//[kbse ];// +  c1over216;(D.f[dirBSE ])[kbs];//kbse
			real mfaca = (D.f[dirBNW])[kbw];//[kbnw ];// +  c1over216;(D.f[dirBNW ])[kbw];//kbnw
											   ////////////////////////////////////////////////////////////////////////////////////
			real drho = ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;

			real rho = c1o1 + drho;
			////////////////////////////////////////////////////////////////////////////////////
			real vvx = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
				(((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
				(mfcbb - mfabb)) / rho;
			real vvy = ((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
				(((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
				(mfbcb - mfbab)) / rho;
			real vvz = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
				(((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
				(mfbbc - mfbba)) / rho;
			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			real fx = forces[0] / (pow((double)c2o1, (double)level)); //zero;//0.0032653/(pow(two,level)); //0.000000005;//(two/1600000.0) / 120.0; //
			real fy = forces[1] / (pow((double)c2o1, (double)level)); //zero;
			real fz = forces[2] / (pow((double)c2o1, (double)level)); //zero;
			vvx += fx*c1o2;
			vvy += fy*c1o2;
			vvz += fz*c1o2;
			////////////////////////////////////////////////////////////////////////////////////
			//real omega = omega_in;
			////////////////////////////////////////////////////////////////////////////////////
			//fast
			real oMdrho = c1o1; // comp special
			real m0, m1, m2;
			real vx2;
			real vy2;
			real vz2;
			vx2 = vvx*vvx;
			vy2 = vvy*vvy;
			vz2 = vvz*vvz;
			////////////////////////////////////////////////////////////////////////////////////
			real wadjust;
			real qudricLimitP = c1o100;// * 0.0001f;
			real qudricLimitM = c1o100;// * 0.0001f;
			real qudricLimitD = c1o100;// * 0.001f;
			//real s9 = minusomega;
			//test
			//s9 = 0.;


			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real EQcbb = c0o1;
			real EQabb = c0o1;
			real EQbcb = c0o1;
			real EQbab = c0o1;
			real EQbbc = c0o1;
			real EQbba = c0o1;
			real EQccb = c0o1;
			real EQaab = c0o1;
			real EQcab = c0o1;
			real EQacb = c0o1;
			real EQcbc = c0o1;
			real EQaba = c0o1;
			real EQcba = c0o1;
			real EQabc = c0o1;
			real EQbcc = c0o1;
			real EQbaa = c0o1;
			real EQbca = c0o1;
			real EQbac = c0o1;
			real EQbbb = c0o1;
			real EQccc = drho * c1o27;
			real EQaac = drho * c1o3;
			real EQcac = drho * c1o9;
			real EQacc = drho * c1o9;
			real EQcca = drho * c1o9;
			real EQaaa = drho;
			real EQcaa = drho * c1o3;
			real EQaca = drho * c1o3;
			////////////////////////////////////////////////////////////////////////////////////
			backwardChimeraWithK(EQaaa, EQaab, EQaac, vvz, vz2, c1o1);
			backwardChimeraWithK(EQaca, EQacb, EQacc, vvz, vz2, c1o3);
			///////////////////////////////////////////////////////////
			EQcaa = EQaca; EQcab = EQacb; EQcac = EQacc;
			///////////////////////////////////////////////////////////
			backwardChimeraWithK(EQcca, EQccb, EQccc, vvz, vz2, c1o9);

			backwardChimeraWithK(EQaaa, EQaba, EQaca, vvy, vy2, c1o6);
			backwardChimeraWithK(EQaab, EQabb, EQacb, vvy, vy2, c2o3);
			backwardChimeraWithK(EQaac, EQabc, EQacc, vvy, vy2, c1o6);
			backwardChimeraWithK(EQcaa, EQcba, EQcca, vvy, vy2, c1o18);
			backwardChimeraWithK(EQcab, EQcbb, EQccb, vvy, vy2, c2o9);
			backwardChimeraWithK(EQcac, EQcbc, EQccc, vvy, vy2, c1o18);

			backwardChimeraWithK(EQaaa, EQbaa, EQcaa, vvx, vx2, c1o36);
			backwardChimeraWithK(EQaab, EQbab, EQcab, vvx, vx2, c1o9);
			backwardChimeraWithK(EQaac, EQbac, EQcac, vvx, vx2, c1o36);
			backwardChimeraWithK(EQaba, EQbba, EQcba, vvx, vx2, c1o9);
			backwardChimeraWithK(EQabb, EQbbb, EQcbb, vvx, vx2, c4o9);
			backwardChimeraWithK(EQabc, EQbbc, EQcbc, vvx, vx2, c1o9);
			backwardChimeraWithK(EQaca, EQbca, EQcca, vvx, vx2, c1o36);
			backwardChimeraWithK(EQacb, EQbcb, EQccb, vvx, vx2, c1o9);
			backwardChimeraWithK(EQacc, EQbcc, EQccc, vvx, vx2, c1o36);

			////////////////////////////////////////////////////////////////////////////////////
			//Pre-condition
			mfcbb -= EQcbb;
			mfabb -= EQabb;
			mfbcb -= EQbcb;
			mfbab -= EQbab;
			mfbbc -= EQbbc;
			mfbba -= EQbba;
			mfccb -= EQccb;
			mfaab -= EQaab;
			mfcab -= EQcab;
			mfacb -= EQacb;
			mfcbc -= EQcbc;
			mfaba -= EQaba;
			mfcba -= EQcba;
			mfabc -= EQabc;
			mfbcc -= EQbcc;
			mfbaa -= EQbaa;
			mfbca -= EQbca;
			mfbac -= EQbac;
			mfbbb -= EQbbb;
			mfccc -= EQccc;
			mfaac -= EQaac;
			mfcac -= EQcac;
			mfacc -= EQacc;
			mfcca -= EQcca;
			mfaaa -= EQaaa;
			mfcaa -= EQcaa;
			mfaca -= EQaca;

			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			forwardChimera(mfaaa, mfaab, mfaac, vvz, vz2);
			forwardChimera(mfaba, mfabb, mfabc, vvz, vz2);
			forwardChimera(mfaca, mfacb, mfacc, vvz, vz2);
			forwardChimera(mfbaa, mfbab, mfbac, vvz, vz2);
			forwardChimera(mfbba, mfbbb, mfbbc, vvz, vz2);
			forwardChimera(mfbca, mfbcb, mfbcc, vvz, vz2);
			forwardChimera(mfcaa, mfcab, mfcac, vvz, vz2);
			forwardChimera(mfcba, mfcbb, mfcbc, vvz, vz2);
			forwardChimera(mfcca, mfccb, mfccc, vvz, vz2);

			forwardChimera(mfaaa, mfaba, mfaca, vvy, vy2);
			forwardChimera(mfaab, mfabb, mfacb, vvy, vy2);
			forwardChimera(mfaac, mfabc, mfacc, vvy, vy2);
			forwardChimera(mfbaa, mfbba, mfbca, vvy, vy2);
			forwardChimera(mfbab, mfbbb, mfbcb, vvy, vy2);
			forwardChimera(mfbac, mfbbc, mfbcc, vvy, vy2);
			forwardChimera(mfcaa, mfcba, mfcca, vvy, vy2);
			forwardChimera(mfcab, mfcbb, mfccb, vvy, vy2);
			forwardChimera(mfcac, mfcbc, mfccc, vvy, vy2);

			forwardChimera(mfaaa, mfbaa, mfcaa, vvx, vx2);
			forwardChimera(mfaab, mfbab, mfcab, vvx, vx2);
			forwardChimera(mfaac, mfbac, mfcac, vvx, vx2);
			forwardChimera(mfaba, mfbba, mfcba, vvx, vx2);
			forwardChimera(mfabb, mfbbb, mfcbb, vvx, vx2);
			forwardChimera(mfabc, mfbbc, mfcbc, vvx, vx2);
			forwardChimera(mfaca, mfbca, mfcca, vvx, vx2);
			forwardChimera(mfacb, mfbcb, mfccb, vvx, vx2);
			forwardChimera(mfacc, mfbcc, mfccc, vvx, vx2);

			//////////////////////////////////////////////////////////////////////////////////////
			////Hin
			//////////////////////////////////////////////////////////////////////////////////////
			//// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// Z - Dir
			//forwardChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2, c4o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// Y - Dir
			//forwardChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2, c1o6);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfaab, mfabb, mfacb, vvy, vy2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c1o18);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2, c2o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfbab, mfbbb, mfbcb, vvy, vy2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2, c2o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2, c1o6);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfcab, mfcbb, mfccb, vvy, vy2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c1o18);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// X - Dir
			//forwardChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, one);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfaba, mfbba, mfcba, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c1o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfaab, mfbab, mfcab, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfabb, mfbbb, mfcbb, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfacb, mfbcb, mfccb, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c1o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfabc, mfbbc, mfcbc, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			real OxxPyyPzz = c1o1; //omega; // one;	//set the bulk viscosity one is high / two is very low and zero is (too) high

			////////////////////////////////////////////////////////////
			//3.
			//////////////////////////////
			real OxyyPxzz = c1o1;
			real OxyyMxzz = c1o1;
			real Oxyz = c1o1;
			////////////////////////////////////////////////////////////
			//4.
			//////////////////////////////
			real O4 = c1o1;
			////////////////////////////////////////////////////////////
			//5.
			//////////////////////////////
			real O5 = c1o1;
			////////////////////////////////////////////////////////////
			//6.
			//////////////////////////////
			real O6 = c1o1;
			////////////////////////////////////////////////////////////


			//central moments to cumulants
			//4.
			real CUMcbb = mfcbb - ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) / rho;
			real CUMbcb = mfbcb - ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) / rho;
			real CUMbbc = mfbbc - ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) / rho;

			real CUMcca = mfcca - (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho - c1o9*(drho / rho));
			real CUMcac = mfcac - (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho - c1o9*(drho / rho));
			real CUMacc = mfacc - (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho - c1o9*(drho / rho));

			//5.
			real CUMbcc = mfbcc - ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) / rho;
			real CUMcbc = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) / rho;
			real CUMccb = mfccb - ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) / rho;

			//6.

			real CUMccc = mfccc + ((-c4o1 *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
				+ (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
					+ c2o1 * (mfcaa * mfaca * mfaac)
					+ c16o1 *  mfbba * mfbab * mfabb) / (rho * rho)
				- c1o3 * (mfacc + mfcac + mfcca) / rho
				- c1o9 * (mfcaa + mfaca + mfaac) / rho
				+ (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
					+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3
				+ c1o27*((drho * drho - drho) / (rho*rho)));




			//2.
			// linear combinations
			real mxxPyyPzz = mfcaa + mfaca + mfaac;
			real mxxMyy = mfcaa - mfaca;
			real mxxMzz = mfcaa - mfaac;

			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			//incl. correction		(hat noch nicht so gut funktioniert...Optimierungsbedarf??)
			{
				real dxux = c1o2 * (-omega) *(mxxMyy + mxxMzz) + c1o2 *  OxxPyyPzz * (mfaaa - mxxPyyPzz);
				real dyuy = dxux + omega * c3o2 * mxxMyy;
				real dzuz = dxux + omega * c3o2 * mxxMzz;

				//relax
				mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz) - c3o1 * (c1o1 - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2 * dzuz);//-magicBulk*OxxPyyPzz;
				mxxMyy += omega * (-mxxMyy) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy);
				mxxMzz += omega * (-mxxMzz) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz);

			}
			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			////no correction
			//mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz);//-magicBulk*OxxPyyPzz;
			//mxxMyy += -(-omega) * (-mxxMyy);
			//mxxMzz += -(-omega) * (-mxxMzz);
			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			mfabb += omega * (-mfabb);
			mfbab += omega * (-mfbab);
			mfbba += omega * (-mfbba);

			//////////////////////////////////////////////////////////////////////////

			// linear combinations back
			mfcaa = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
			mfaca = c1o3 * (-c2o1*  mxxMyy + mxxMzz + mxxPyyPzz);
			mfaac = c1o3 * (mxxMyy - c2o1* mxxMzz + mxxPyyPzz);

			//3.
			// linear combinations

			real mxxyPyzz = mfcba + mfabc;
			real mxxyMyzz = mfcba - mfabc;

			real mxxzPyyz = mfcab + mfacb;
			real mxxzMyyz = mfcab - mfacb;

			real mxyyPxzz = mfbca + mfbac;
			real mxyyMxzz = mfbca - mfbac;

			//relax
			//////////////////////////////////////////////////////////////////////////
			mfbbb += OxyyMxzz * (-mfbbb);
			mxxyPyzz += OxyyPxzz * (-mxxyPyzz);
			mxxyMyzz += OxyyMxzz * (-mxxyMyzz);
			mxxzPyyz += OxyyPxzz * (-mxxzPyyz);
			mxxzMyyz += OxyyMxzz * (-mxxzMyyz);
			mxyyPxzz += OxyyPxzz * (-mxyyPxzz);
			mxyyMxzz += OxyyMxzz * (-mxyyMxzz);
			//////////////////////////////////////////////////////////////////////////

			mfcba = (mxxyMyzz + mxxyPyzz) * c1o2;
			mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
			mfcab = (mxxzMyyz + mxxzPyyz) * c1o2;
			mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
			mfbca = (mxyyMxzz + mxyyPxzz) * c1o2;
			mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;

			//4.
			//////////////////////////////////////////////////////////////////////////
			CUMacc += O4 * (-CUMacc);
			CUMcac += O4 * (-CUMcac);
			CUMcca += O4 * (-CUMcca);

			CUMbbc += O4 * (-CUMbbc);
			CUMbcb += O4 * (-CUMbcb);
			CUMcbb += O4 * (-CUMcbb);
			//////////////////////////////////////////////////////////////////////////


			//5.
			CUMbcc += O5 * (-CUMbcc);
			CUMcbc += O5 * (-CUMcbc);
			CUMccb += O5 * (-CUMccb);

			//6.
			CUMccc += O6 * (-CUMccc);



			//back cumulants to central moments
			//4.
			mfcbb = CUMcbb + ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) / rho;
			mfbcb = CUMbcb + ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) / rho;
			mfbbc = CUMbbc + ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) / rho;

			mfcca = CUMcca + (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho - c1o9*(drho / rho));
			mfcac = CUMcac + (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho - c1o9*(drho / rho));
			mfacc = CUMacc + (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho - c1o9*(drho / rho));

			//5.
			mfbcc = CUMbcc + ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) / rho;
			mfcbc = CUMcbc + ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) / rho;
			mfccb = CUMccb + ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) / rho;

			//6.
			mfccc = CUMccc - ((-c4o1 *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
				+ (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
					+ c2o1 * (mfcaa * mfaca * mfaac)
					+ c16o1 *  mfbba * mfbab * mfabb) / (rho * rho)
				- c1o3 * (mfacc + mfcac + mfcca) / rho
				- c1o9 * (mfcaa + mfaca + mfaac) / rho
				+ (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
					+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3
				+ c1o27*((drho * drho - drho) / (rho*rho)));

			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			mfbaa = -mfbaa;
			mfaba = -mfaba;
			mfaab = -mfaab;
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			//back
			////////////////////////////////////////////////////////////////////////////////////
			backwardChimera(mfaaa, mfaab, mfaac, vvz, vz2);
			backwardChimera(mfaba, mfabb, mfabc, vvz, vz2);
			backwardChimera(mfaca, mfacb, mfacc, vvz, vz2);
			backwardChimera(mfbaa, mfbab, mfbac, vvz, vz2);
			backwardChimera(mfbba, mfbbb, mfbbc, vvz, vz2);
			backwardChimera(mfbca, mfbcb, mfbcc, vvz, vz2);
			backwardChimera(mfcaa, mfcab, mfcac, vvz, vz2);
			backwardChimera(mfcba, mfcbb, mfcbc, vvz, vz2);
			backwardChimera(mfcca, mfccb, mfccc, vvz, vz2);

			backwardChimera(mfaaa, mfaba, mfaca, vvy, vy2);
			backwardChimera(mfaab, mfabb, mfacb, vvy, vy2);
			backwardChimera(mfaac, mfabc, mfacc, vvy, vy2);
			backwardChimera(mfbaa, mfbba, mfbca, vvy, vy2);
			backwardChimera(mfbab, mfbbb, mfbcb, vvy, vy2);
			backwardChimera(mfbac, mfbbc, mfbcc, vvy, vy2);
			backwardChimera(mfcaa, mfcba, mfcca, vvy, vy2);
			backwardChimera(mfcab, mfcbb, mfccb, vvy, vy2);
			backwardChimera(mfcac, mfcbc, mfccc, vvy, vy2);

			backwardChimera(mfaaa, mfbaa, mfcaa, vvx, vx2);
			backwardChimera(mfaab, mfbab, mfcab, vvx, vx2);
			backwardChimera(mfaac, mfbac, mfcac, vvx, vx2);
			backwardChimera(mfaba, mfbba, mfcba, vvx, vx2);
			backwardChimera(mfabb, mfbbb, mfcbb, vvx, vx2);
			backwardChimera(mfabc, mfbbc, mfcbc, vvx, vx2);
			backwardChimera(mfaca, mfbca, mfcca, vvx, vx2);
			backwardChimera(mfacb, mfbcb, mfccb, vvx, vx2);
			backwardChimera(mfacc, mfbcc, mfccc, vvx, vx2);

			////////////////////////////////////////////////////////////////////////////////////
			//mfcbb += EQcbb;
			//mfabb += EQabb;
			//mfbcb += EQbcb;
			//mfbab += EQbab;
			//mfbbc += EQbbc;
			//mfbba += EQbba;
			//mfccb += EQccb;
			//mfaab += EQaab;
			//mfcab += EQcab;
			//mfacb += EQacb;
			//mfcbc += EQcbc;
			//mfaba += EQaba;
			//mfcba += EQcba;
			//mfabc += EQabc;
			//mfbcc += EQbcc;
			//mfbaa += EQbaa;
			//mfbca += EQbca;
			//mfbac += EQbac;
			//mfbbb += EQbbb;
			//mfccc += EQccc;
			//mfaac += EQaac;
			//mfcac += EQcac;
			//mfacc += EQacc;
			//mfcca += EQcca;
			//mfaaa += EQaaa;
			//mfcaa += EQcaa;
			//mfaca += EQaca;
			////////////////////////////////////////////////////////////////////////////////////
			////Error diffusion
			real fTEMP = mfbbb + EQbbb;
			real delta0 = mfbbb - (fTEMP - EQbbb);
			delta0 *= c1o4;
			mfbbb = fTEMP;


			fTEMP = mfcbb + EQcbb;
			real deltacbb = mfcbb - (fTEMP - EQcbb);
			mfcbb = fTEMP;
			//mfcbb+=EQcbb;

			fTEMP = mfabb + EQabb;
			real deltaabb = mfabb - (fTEMP - EQabb);
			mfabb = fTEMP;
			//mfabb+=EQabb;

			fTEMP = mfbcb + EQbcb;
			real deltabcb = mfbcb - (fTEMP - EQbcb);
			mfbcb = fTEMP;
			//mfbcb+=EQbcb;

			fTEMP = mfbab + EQbab;
			real deltabab = mfbab - (fTEMP - EQbab);
			mfbab = fTEMP;
			//mfbab+=EQbab;

			fTEMP = mfbbc + EQbbc;
			real deltabbc = mfbbc - (fTEMP - EQbbc);
			mfbbc = fTEMP;
			//mfbbc+=EQbbc;

			fTEMP = mfbba + EQbba;
			real deltabba = mfbba - (fTEMP - EQbba);
			mfbba = fTEMP;
			//mfbba+=EQbba;

			EQccb += (delta0 + c1o2*(deltacbb + deltabcb));
			fTEMP = mfccb + EQccb;
			real deltaccb = mfccb - (fTEMP - EQccb);
			mfccb = fTEMP;
			//mfccb+=EQccb+(delta0+c1o2*(deltacbb+deltabcb));

			EQaab += (delta0 + c1o2*(deltaabb + deltabab));
			fTEMP = mfaab + EQaab;
			real deltaaab = mfaab - (fTEMP - EQaab);
			mfaab = fTEMP;
			//mfaab+=EQaab+(delta0+c1o2*(deltaabb+deltabab));

			EQcab += (delta0 + c1o2*(deltacbb + deltabab));
			fTEMP = mfcab + EQcab;
			real deltacab = mfcab - (fTEMP - EQcab);
			mfcab = fTEMP;
			//mfcab+=EQcab+(delta0+c1o2*(deltacbb+deltabab));

			EQacb += (delta0 + c1o2*(deltaabb + deltabcb));
			fTEMP = mfacb + EQacb;
			real deltaacb = mfacb - (fTEMP - EQacb);
			mfacb = fTEMP;
			//mfacb+=EQacb+(delta0+c1o2*(deltaabb+deltabcb));

			EQcbc += (delta0 + c1o2*(deltacbb + deltabbc));
			fTEMP = mfcbc + EQcbc;
			real deltacbc = mfcbc - (fTEMP - EQcbc);
			mfcbc = fTEMP;
			//mfcbc+=EQcbc+(delta0+c1o2*(deltacbb+deltabbc));

			EQaba += (delta0 + c1o2*(deltaabb + deltabba));
			fTEMP = mfaba + EQaba;
			real deltaaba = mfaba - (fTEMP - EQaba);
			mfaba = fTEMP;
			//mfaba+=EQaba+(delta0+c1o2*(deltaabb+deltabba));

			EQcba += (delta0 + c1o2*(deltacbb + deltabba));
			fTEMP = mfcba + EQcba;
			real deltacba = mfcba - (fTEMP - EQcba);
			mfcba = fTEMP;
			//mfcba+=EQcba+(delta0+c1o2*(deltacbb+deltabba));

			EQabc += (delta0 + c1o2*(deltaabb + deltabbc));
			fTEMP = mfabc + EQabc;
			real deltaabc = mfabc - (fTEMP - EQabc);
			mfabc = fTEMP;
			//mfabc+=EQabc+(delta0+c1o2*(deltaabb+deltabbc));

			EQbcc += (delta0 + c1o2*(deltabcb + deltabbc));
			fTEMP = mfbcc + EQbcc;
			real deltabcc = mfbcc - (fTEMP - EQbcc);
			mfbcc = fTEMP;
			//mfbcc+=EQbcc+(delta0+c1o2*(deltabcb+deltabbc));

			EQbaa += (delta0 + c1o2*(deltabab + deltabba));
			fTEMP = mfbaa + EQbaa;
			real deltabaa = mfbaa - (fTEMP - EQbaa);
			mfbaa = fTEMP;
			//mfbaa+=EQbaa+(delta0+c1o2*(deltabab+deltabba));

			EQbca += (delta0 + c1o2*(deltabcb + deltabba));
			fTEMP = mfbca + EQbca;
			real deltabca = mfbca - (fTEMP - EQbca);
			mfbca = fTEMP;
			//mfbca+=EQbca+(delta0+c1o2*(deltabcb+deltabba));

			EQbac += (delta0 + c1o2*(deltabab + deltabbc));
			fTEMP = mfbac + EQbac;
			real deltabac = mfbac - (fTEMP - EQbac);
			mfbac = fTEMP;
			//mfbac+=EQbac+(delta0+c1o2*(deltabab+deltabbc));

			mfccc += EQccc - (delta0 + c1o4*(deltacbb + deltabcb + deltabbc) - c1o2*(deltabcc + deltacbc + deltaccb));
			mfaac += EQaac - (delta0 + c1o4*(deltaabb + deltabab + deltabbc) - c1o2*(deltabac + deltaabc + deltaaab));
			mfcac += EQcac - (delta0 + c1o4*(deltacbb + deltabab + deltabbc) - c1o2*(deltabac + deltacbc + deltacab));
			mfacc += EQacc - (delta0 + c1o4*(deltaabb + deltabcb + deltabbc) - c1o2*(deltabcc + deltaabc + deltaacb));
			mfcca += EQcca - (delta0 + c1o4*(deltacbb + deltabcb + deltabba) - c1o2*(deltabca + deltacba + deltaccb));
			mfaaa += EQaaa - (delta0 + c1o4*(deltaabb + deltabab + deltabba) - c1o2*(deltabaa + deltaaba + deltaaab));
			mfcaa += EQcaa - (delta0 + c1o4*(deltacbb + deltabab + deltabba) - c1o2*(deltabaa + deltacba + deltacab));
			mfaca += EQaca - (delta0 + c1o4*(deltaabb + deltabcb + deltabba) - c1o2*(deltabca + deltaaba + deltaacb));



			//////////////////////////////////////////////////////////////////////////////////////
			////back
			//////////////////////////////////////////////////////////////////////////////////////
			////mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// Z - Dir
			//backwardChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, one);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfaba, mfabb, mfabc, vvz, vz2);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c1o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbaa, mfbab, mfbac, vvz, vz2);
			///////////b//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbba, mfbbb, mfbbc, vvz, vz2);
			///////////b//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbca, mfbcb, mfbcc, vvz, vz2);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c1o3);
			///////////c//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfcba, mfcbb, mfcbc, vvz, vz2);
			///////////c//////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			////mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// Y - Dir
			//backwardChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2, c1o6);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaab, mfabb, mfacb, vvy, vy2, c2o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c1o6);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbaa, mfbba, mfbca, vvz, vz2);
			///////////b//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbab, mfbbb, mfbcb, vvz, vz2);
			///////////b//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbac, mfbbc, mfbcc, vvz, vz2);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2, c1o18);
			///////////c//////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcab, mfcbb, mfccb, vvy, vy2, c2o9);
			///////////c//////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c1o18);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			////mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// X - Dir
			//backwardChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaba, mfbba, mfcba, vvx, vx2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaab, mfbab, mfcab, vvx, vx2, c1o9);
			/////////////b////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfabb, mfbbb, mfcbb, vvx, vx2, c4o9);
			/////////////b////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfacb, mfbcb, mfccb, vvx, vx2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c1o36);
			/////////////c////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfabc, mfbbc, mfcbc, vvx, vx2, c1o9);
			/////////////c////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////

			////////////////////////////////////////////////////////////////////////////////////////
			//real drhoPost =
			//	((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
			//	(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
			//		((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;
			//mfbbb += drho - drhoPost;
			////////////////////////////////////////////////////////////////////////////////////
			(D.f[dirE])[k] = mfabb;//(D.f[ dirE   ])[ke   ] = mfabb;// -  c2over27 ;  (D.f[ dirE   ])[k   ]                                                                     
			(D.f[dirW])[kw] = mfcbb;//(D.f[ dirW   ])[kw   ] = mfcbb;// -  c2over27 ;  (D.f[ dirW   ])[kw  ]                                                                   
			(D.f[dirN])[k] = mfbab;//(D.f[ dirN   ])[kn   ] = mfbab;// -  c2over27 ;	 (D.f[ dirN   ])[k   ]
			(D.f[dirS])[ks] = mfbcb;//(D.f[ dirS   ])[ks   ] = mfbcb;// -  c2over27 ;	 (D.f[ dirS   ])[ks  ]
			(D.f[dirT])[k] = mfbba;//(D.f[ dirT   ])[kt   ] = mfbba;// -  c2over27 ;	 (D.f[ dirT   ])[k   ]
			(D.f[dirB])[kb] = mfbbc;//(D.f[ dirB   ])[kb   ] = mfbbc;// -  c2over27 ;	 (D.f[ dirB   ])[kb  ]
			(D.f[dirNE])[k] = mfaab;//(D.f[ dirNE  ])[kne  ] = mfaab;// -  c1over54 ;	 (D.f[ dirNE  ])[k   ]
			(D.f[dirSW])[ksw] = mfccb;//(D.f[ dirSW  ])[ksw  ] = mfccb;// -  c1over54 ;	 (D.f[ dirSW  ])[ksw ]
			(D.f[dirSE])[ks] = mfacb;//(D.f[ dirSE  ])[kse  ] = mfacb;// -  c1over54 ;	 (D.f[ dirSE  ])[ks  ]
			(D.f[dirNW])[kw] = mfcab;//(D.f[ dirNW  ])[knw  ] = mfcab;// -  c1over54 ;	 (D.f[ dirNW  ])[kw  ]
			(D.f[dirTE])[k] = mfaba;//(D.f[ dirTE  ])[kte  ] = mfaba;// -  c1over54 ;	 (D.f[ dirTE  ])[k   ]
			(D.f[dirBW])[kbw] = mfcbc;//(D.f[ dirBW  ])[kbw  ] = mfcbc;// -  c1over54 ;	 (D.f[ dirBW  ])[kbw ]
			(D.f[dirBE])[kb] = mfabc;//(D.f[ dirBE  ])[kbe  ] = mfabc;// -  c1over54 ;	 (D.f[ dirBE  ])[kb  ]
			(D.f[dirTW])[kw] = mfcba;//(D.f[ dirTW  ])[ktw  ] = mfcba;// -  c1over54 ;	 (D.f[ dirTW  ])[kw  ]
			(D.f[dirTN])[k] = mfbaa;//(D.f[ dirTN  ])[ktn  ] = mfbaa;// -  c1over54 ;	 (D.f[ dirTN  ])[k   ]
			(D.f[dirBS])[kbs] = mfbcc;//(D.f[ dirBS  ])[kbs  ] = mfbcc;// -  c1over54 ;	 (D.f[ dirBS  ])[kbs ]
			(D.f[dirBN])[kb] = mfbac;//(D.f[ dirBN  ])[kbn  ] = mfbac;// -  c1over54 ;	 (D.f[ dirBN  ])[kb  ]
			(D.f[dirTS])[ks] = mfbca;//(D.f[ dirTS  ])[kts  ] = mfbca;// -  c1over54 ;	 (D.f[ dirTS  ])[ks  ]
			(D.f[dirZERO])[k] = mfbbb;//(D.f[ dirZERO])[kzero] = mfbbb;// -  c8over27 ;	 (D.f[ dirZERO])[k   ]
			(D.f[dirTNE])[k] = mfaaa;//(D.f[ dirTNE ])[ktne ] = mfaaa;// -  c1over216;	 (D.f[ dirTNE ])[k   ]
			(D.f[dirTSE])[ks] = mfaca;//(D.f[ dirTSE ])[ktse ] = mfaca;// -  c1over216;	 (D.f[ dirTSE ])[ks  ]
			(D.f[dirBNE])[kb] = mfaac;//(D.f[ dirBNE ])[kbne ] = mfaac;// -  c1over216;	 (D.f[ dirBNE ])[kb  ]
			(D.f[dirBSE])[kbs] = mfacc;//(D.f[ dirBSE ])[kbse ] = mfacc;// -  c1over216;	 (D.f[ dirBSE ])[kbs ]
			(D.f[dirTNW])[kw] = mfcaa;//(D.f[ dirTNW ])[ktnw ] = mfcaa;// -  c1over216;	 (D.f[ dirTNW ])[kw  ]
			(D.f[dirTSW])[ksw] = mfcca;//(D.f[ dirTSW ])[ktsw ] = mfcca;// -  c1over216;	 (D.f[ dirTSW ])[ksw ]
			(D.f[dirBNW])[kbw] = mfcac;//(D.f[ dirBNW ])[kbnw ] = mfcac;// -  c1over216;	 (D.f[ dirBNW ])[kbw ]
			(D.f[dirBSW])[kbsw] = mfccc;//(D.f[ dirBSW ])[kbsw ] = mfccc;// -  c1over216;	 (D.f[ dirBSW ])[kbsw]
										////////////////////////////////////////////////////////////////////////////////////
		}
	}
}
////////////////////////////////////////////////////////////////////////////////








































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void Cumulant_One_preconditioned_chim_Comp_SP_27(
	real omega,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* DDStart,
	int size_Mat,
	int level,
	real* forces,
	bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if (BC >= GEO_FLUID/*(BC != GEO_SOLID) && (BC != GEO_VOID)*/)
		{
			Distributions27 D;
			if (EvenOrOdd == true)
			{
				D.f[dirE] = &DDStart[dirE   *size_Mat];
				D.f[dirW] = &DDStart[dirW   *size_Mat];
				D.f[dirN] = &DDStart[dirN   *size_Mat];
				D.f[dirS] = &DDStart[dirS   *size_Mat];
				D.f[dirT] = &DDStart[dirT   *size_Mat];
				D.f[dirB] = &DDStart[dirB   *size_Mat];
				D.f[dirNE] = &DDStart[dirNE  *size_Mat];
				D.f[dirSW] = &DDStart[dirSW  *size_Mat];
				D.f[dirSE] = &DDStart[dirSE  *size_Mat];
				D.f[dirNW] = &DDStart[dirNW  *size_Mat];
				D.f[dirTE] = &DDStart[dirTE  *size_Mat];
				D.f[dirBW] = &DDStart[dirBW  *size_Mat];
				D.f[dirBE] = &DDStart[dirBE  *size_Mat];
				D.f[dirTW] = &DDStart[dirTW  *size_Mat];
				D.f[dirTN] = &DDStart[dirTN  *size_Mat];
				D.f[dirBS] = &DDStart[dirBS  *size_Mat];
				D.f[dirBN] = &DDStart[dirBN  *size_Mat];
				D.f[dirTS] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirTNE] = &DDStart[dirTNE *size_Mat];
				D.f[dirTSW] = &DDStart[dirTSW *size_Mat];
				D.f[dirTSE] = &DDStart[dirTSE *size_Mat];
				D.f[dirTNW] = &DDStart[dirTNW *size_Mat];
				D.f[dirBNE] = &DDStart[dirBNE *size_Mat];
				D.f[dirBSW] = &DDStart[dirBSW *size_Mat];
				D.f[dirBSE] = &DDStart[dirBSE *size_Mat];
				D.f[dirBNW] = &DDStart[dirBNW *size_Mat];
			}
			else
			{
				D.f[dirW] = &DDStart[dirE   *size_Mat];
				D.f[dirE] = &DDStart[dirW   *size_Mat];
				D.f[dirS] = &DDStart[dirN   *size_Mat];
				D.f[dirN] = &DDStart[dirS   *size_Mat];
				D.f[dirB] = &DDStart[dirT   *size_Mat];
				D.f[dirT] = &DDStart[dirB   *size_Mat];
				D.f[dirSW] = &DDStart[dirNE  *size_Mat];
				D.f[dirNE] = &DDStart[dirSW  *size_Mat];
				D.f[dirNW] = &DDStart[dirSE  *size_Mat];
				D.f[dirSE] = &DDStart[dirNW  *size_Mat];
				D.f[dirBW] = &DDStart[dirTE  *size_Mat];
				D.f[dirTE] = &DDStart[dirBW  *size_Mat];
				D.f[dirTW] = &DDStart[dirBE  *size_Mat];
				D.f[dirBE] = &DDStart[dirTW  *size_Mat];
				D.f[dirBS] = &DDStart[dirTN  *size_Mat];
				D.f[dirTN] = &DDStart[dirBS  *size_Mat];
				D.f[dirTS] = &DDStart[dirBN  *size_Mat];
				D.f[dirBN] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirBSW] = &DDStart[dirTNE *size_Mat];
				D.f[dirBNE] = &DDStart[dirTSW *size_Mat];
				D.f[dirBNW] = &DDStart[dirTSE *size_Mat];
				D.f[dirBSE] = &DDStart[dirTNW *size_Mat];
				D.f[dirTSW] = &DDStart[dirBNE *size_Mat];
				D.f[dirTNE] = &DDStart[dirBSW *size_Mat];
				D.f[dirTNW] = &DDStart[dirBSE *size_Mat];
				D.f[dirTSE] = &DDStart[dirBNW *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			//unsigned int kzero= k;
			//unsigned int ke   = k;
			unsigned int kw = neighborX[k];
			//unsigned int kn   = k;
			unsigned int ks = neighborY[k];
			//unsigned int kt   = k;
			unsigned int kb = neighborZ[k];
			unsigned int ksw = neighborY[kw];
			//unsigned int kne  = k;
			//unsigned int kse  = ks;
			//unsigned int knw  = kw;
			unsigned int kbw = neighborZ[kw];
			//unsigned int kte  = k;
			//unsigned int kbe  = kb;
			//unsigned int ktw  = kw;
			unsigned int kbs = neighborZ[ks];
			//unsigned int ktn  = k;
			//unsigned int kbn  = kb;
			//unsigned int kts  = ks;
			//unsigned int ktse = ks;
			//unsigned int kbnw = kbw;
			//unsigned int ktnw = kw;
			//unsigned int kbse = kbs;
			//unsigned int ktsw = ksw;
			//unsigned int kbne = kb;
			//unsigned int ktne = k;
			unsigned int kbsw = neighborZ[ksw];



			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[dirE])[k];//[ke   ];// +  c2over27 ;(D.f[dirE   ])[k  ];//ke
			real mfabb = (D.f[dirW])[kw];//[kw   ];// +  c2over27 ;(D.f[dirW   ])[kw ];
			real mfbcb = (D.f[dirN])[k];//[kn   ];// +  c2over27 ;(D.f[dirN   ])[k  ];//kn
			real mfbab = (D.f[dirS])[ks];//[ks   ];// +  c2over27 ;(D.f[dirS   ])[ks ];
			real mfbbc = (D.f[dirT])[k];//[kt   ];// +  c2over27 ;(D.f[dirT   ])[k  ];//kt
			real mfbba = (D.f[dirB])[kb];//[kb   ];// +  c2over27 ;(D.f[dirB   ])[kb ];
			real mfccb = (D.f[dirNE])[k];//[kne  ];// +  c1over54 ;(D.f[dirNE  ])[k  ];//kne
			real mfaab = (D.f[dirSW])[ksw];//[ksw  ];// +  c1over54 ;(D.f[dirSW  ])[ksw];
			real mfcab = (D.f[dirSE])[ks];//[kse  ];// +  c1over54 ;(D.f[dirSE  ])[ks ];//kse
			real mfacb = (D.f[dirNW])[kw];//[knw  ];// +  c1over54 ;(D.f[dirNW  ])[kw ];//knw
			real mfcbc = (D.f[dirTE])[k];//[kte  ];// +  c1over54 ;(D.f[dirTE  ])[k  ];//kte
			real mfaba = (D.f[dirBW])[kbw];//[kbw  ];// +  c1over54 ;(D.f[dirBW  ])[kbw];
			real mfcba = (D.f[dirBE])[kb];//[kbe  ];// +  c1over54 ;(D.f[dirBE  ])[kb ];//kbe
			real mfabc = (D.f[dirTW])[kw];//[ktw  ];// +  c1over54 ;(D.f[dirTW  ])[kw ];//ktw
			real mfbcc = (D.f[dirTN])[k];//[ktn  ];// +  c1over54 ;(D.f[dirTN  ])[k  ];//ktn
			real mfbaa = (D.f[dirBS])[kbs];//[kbs  ];// +  c1over54 ;(D.f[dirBS  ])[kbs];
			real mfbca = (D.f[dirBN])[kb];//[kbn  ];// +  c1over54 ;(D.f[dirBN  ])[kb ];//kbn
			real mfbac = (D.f[dirTS])[ks];//[kts  ];// +  c1over54 ;(D.f[dirTS  ])[ks ];//kts
			real mfbbb = (D.f[dirZERO])[k];//[kzero];// +  c8over27 ;(D.f[dirZERO])[k  ];//kzero
			real mfccc = (D.f[dirTNE])[k];//[ktne ];// +  c1over216;(D.f[dirTNE ])[k  ];//ktne
			real mfaac = (D.f[dirTSW])[ksw];//[ktsw ];// +  c1over216;(D.f[dirTSW ])[ksw];//ktsw
			real mfcac = (D.f[dirTSE])[ks];//[ktse ];// +  c1over216;(D.f[dirTSE ])[ks ];//ktse
			real mfacc = (D.f[dirTNW])[kw];//[ktnw ];// +  c1over216;(D.f[dirTNW ])[kw ];//ktnw
			real mfcca = (D.f[dirBNE])[kb];//[kbne ];// +  c1over216;(D.f[dirBNE ])[kb ];//kbne
			real mfaaa = (D.f[dirBSW])[kbsw];//[kbsw ];// +  c1over216;(D.f[dirBSW ])[kbsw];
			real mfcaa = (D.f[dirBSE])[kbs];//[kbse ];// +  c1over216;(D.f[dirBSE ])[kbs];//kbse
			real mfaca = (D.f[dirBNW])[kbw];//[kbnw ];// +  c1over216;(D.f[dirBNW ])[kbw];//kbnw
											   ////////////////////////////////////////////////////////////////////////////////////
			real drho = ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;

			real rho = c1o1 + drho;
			////////////////////////////////////////////////////////////////////////////////////
			real vvx = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
				(((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
				(mfcbb - mfabb)) / rho;
			real vvy = ((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
				(((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
				(mfbcb - mfbab)) / rho;
			real vvz = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
				(((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
				(mfbbc - mfbba)) / rho;
			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			real fx = forces[0] / (pow((double)c2o1, (double)level)); //zero;//0.0032653/(pow(two,level)); //0.000000005;//(two/1600000.0) / 120.0; //
			real fy = forces[1] / (pow((double)c2o1, (double)level)); //zero;
			real fz = forces[2] / (pow((double)c2o1, (double)level)); //zero;
			vvx += fx*c1o2;
			vvy += fy*c1o2;
			vvz += fz*c1o2;
			////////////////////////////////////////////////////////////////////////////////////
			//real omega = omega_in;
			////////////////////////////////////////////////////////////////////////////////////
			//fast
			real oMdrho = c1o1; // comp special
			real m0, m1, m2;
			real vx2;
			real vy2;
			real vz2;
			vx2 = vvx*vvx;
			vy2 = vvy*vvy;
			vz2 = vvz*vvz;
			////////////////////////////////////////////////////////////////////////////////////
			real wadjust;
			real qudricLimitP = c1o100;// * 0.0001f;
			real qudricLimitM = c1o100;// * 0.0001f;
			real qudricLimitD = c1o100;// * 0.001f;
			//real s9 = minusomega;
			//test
			//s9 = 0.;


			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real EQcbb = c0o1;
			real EQabb = c0o1;
			real EQbcb = c0o1;
			real EQbab = c0o1;
			real EQbbc = c0o1;
			real EQbba = c0o1;
			real EQccb = c0o1;
			real EQaab = c0o1;
			real EQcab = c0o1;
			real EQacb = c0o1;
			real EQcbc = c0o1;
			real EQaba = c0o1;
			real EQcba = c0o1;
			real EQabc = c0o1;
			real EQbcc = c0o1;
			real EQbaa = c0o1;
			real EQbca = c0o1;
			real EQbac = c0o1;
			real EQbbb = c0o1;
			real EQccc = drho * c1o27;
			real EQaac = drho * c1o3;
			real EQcac = drho * c1o9;
			real EQacc = drho * c1o9;
			real EQcca = drho * c1o9;
			real EQaaa = drho;
			real EQcaa = drho * c1o3;
			real EQaca = drho * c1o3;
			////////////////////////////////////////////////////////////////////////////////////
			backwardChimeraWithK(EQaaa, EQaab, EQaac, vvz, vz2, c1o1);
			backwardChimeraWithK(EQaca, EQacb, EQacc, vvz, vz2, c1o3);
			///////////////////////////////////////////////////////////
			EQcaa = EQaca; EQcab = EQacb; EQcac = EQacc;
			///////////////////////////////////////////////////////////
			backwardChimeraWithK(EQcca, EQccb, EQccc, vvz, vz2, c1o9);

			backwardChimeraWithK(EQaaa, EQaba, EQaca, vvy, vy2, c1o6);
			backwardChimeraWithK(EQaab, EQabb, EQacb, vvy, vy2, c2o3);
			backwardChimeraWithK(EQaac, EQabc, EQacc, vvy, vy2, c1o6);
			backwardChimeraWithK(EQcaa, EQcba, EQcca, vvy, vy2, c1o18);
			backwardChimeraWithK(EQcab, EQcbb, EQccb, vvy, vy2, c2o9);
			backwardChimeraWithK(EQcac, EQcbc, EQccc, vvy, vy2, c1o18);

			backwardChimeraWithK(EQaaa, EQbaa, EQcaa, vvx, vx2, c1o36);
			backwardChimeraWithK(EQaab, EQbab, EQcab, vvx, vx2, c1o9);
			backwardChimeraWithK(EQaac, EQbac, EQcac, vvx, vx2, c1o36);
			backwardChimeraWithK(EQaba, EQbba, EQcba, vvx, vx2, c1o9);
			backwardChimeraWithK(EQabb, EQbbb, EQcbb, vvx, vx2, c4o9);
			backwardChimeraWithK(EQabc, EQbbc, EQcbc, vvx, vx2, c1o9);
			backwardChimeraWithK(EQaca, EQbca, EQcca, vvx, vx2, c1o36);
			backwardChimeraWithK(EQacb, EQbcb, EQccb, vvx, vx2, c1o9);
			backwardChimeraWithK(EQacc, EQbcc, EQccc, vvx, vx2, c1o36);

			////////////////////////////////////////////////////////////////////////////////////
			//Pre-condition
			mfcbb -= EQcbb;
			mfabb -= EQabb;
			mfbcb -= EQbcb;
			mfbab -= EQbab;
			mfbbc -= EQbbc;
			mfbba -= EQbba;
			mfccb -= EQccb;
			mfaab -= EQaab;
			mfcab -= EQcab;
			mfacb -= EQacb;
			mfcbc -= EQcbc;
			mfaba -= EQaba;
			mfcba -= EQcba;
			mfabc -= EQabc;
			mfbcc -= EQbcc;
			mfbaa -= EQbaa;
			mfbca -= EQbca;
			mfbac -= EQbac;
			mfbbb -= EQbbb;
			mfccc -= EQccc;
			mfaac -= EQaac;
			mfcac -= EQcac;
			mfacc -= EQacc;
			mfcca -= EQcca;
			mfaaa -= EQaaa;
			mfcaa -= EQcaa;
			mfaca -= EQaca;

			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			forwardChimera(mfaaa, mfaab, mfaac, vvz, vz2);
			forwardChimera(mfaba, mfabb, mfabc, vvz, vz2);
			forwardChimera(mfaca, mfacb, mfacc, vvz, vz2);
			forwardChimera(mfbaa, mfbab, mfbac, vvz, vz2);
			forwardChimera(mfbba, mfbbb, mfbbc, vvz, vz2);
			forwardChimera(mfbca, mfbcb, mfbcc, vvz, vz2);
			forwardChimera(mfcaa, mfcab, mfcac, vvz, vz2);
			forwardChimera(mfcba, mfcbb, mfcbc, vvz, vz2);
			forwardChimera(mfcca, mfccb, mfccc, vvz, vz2);

			forwardChimera(mfaaa, mfaba, mfaca, vvy, vy2);
			forwardChimera(mfaab, mfabb, mfacb, vvy, vy2);
			forwardChimera(mfaac, mfabc, mfacc, vvy, vy2);
			forwardChimera(mfbaa, mfbba, mfbca, vvy, vy2);
			forwardChimera(mfbab, mfbbb, mfbcb, vvy, vy2);
			forwardChimera(mfbac, mfbbc, mfbcc, vvy, vy2);
			forwardChimera(mfcaa, mfcba, mfcca, vvy, vy2);
			forwardChimera(mfcab, mfcbb, mfccb, vvy, vy2);
			forwardChimera(mfcac, mfcbc, mfccc, vvy, vy2);

			forwardChimera(mfaaa, mfbaa, mfcaa, vvx, vx2);
			forwardChimera(mfaab, mfbab, mfcab, vvx, vx2);
			forwardChimera(mfaac, mfbac, mfcac, vvx, vx2);
			forwardChimera(mfaba, mfbba, mfcba, vvx, vx2);
			forwardChimera(mfabb, mfbbb, mfcbb, vvx, vx2);
			forwardChimera(mfabc, mfbbc, mfcbc, vvx, vx2);
			forwardChimera(mfaca, mfbca, mfcca, vvx, vx2);
			forwardChimera(mfacb, mfbcb, mfccb, vvx, vx2);
			forwardChimera(mfacc, mfbcc, mfccc, vvx, vx2);

			//////////////////////////////////////////////////////////////////////////////////////
			////Hin
			//////////////////////////////////////////////////////////////////////////////////////
			//// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// Z - Dir
			//forwardChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2, c4o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// Y - Dir
			//forwardChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2, c1o6);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfaab, mfabb, mfacb, vvy, vy2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c1o18);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2, c2o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfbab, mfbbb, mfbcb, vvy, vy2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2, c2o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2, c1o6);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfcab, mfcbb, mfccb, vvy, vy2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c1o18);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// X - Dir
			//forwardChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, one);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfaba, mfbba, mfcba, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c1o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfaab, mfbab, mfcab, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfabb, mfbbb, mfcbb, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfacb, mfbcb, mfccb, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c1o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimera(mfabc, mfbbc, mfcbc, vvx, vx2);
			//////////////////////////////////////////////////////////////////////////////////////
			//forwardChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			real OxxPyyPzz = c1o1; //omega; // one;	//set the bulk viscosity one is high / two is very low and zero is (too) high

			////////////////////////////////////////////////////////////
			//3.
			//////////////////////////////
			real OxyyPxzz = c1o1;
			real OxyyMxzz = c1o1;
			real Oxyz = c1o1;
			////////////////////////////////////////////////////////////
			//4.
			//////////////////////////////
			real O4 = c1o1;
			////////////////////////////////////////////////////////////
			//5.
			//////////////////////////////
			real O5 = c1o1;
			////////////////////////////////////////////////////////////
			//6.
			//////////////////////////////
			real O6 = c1o1;
			////////////////////////////////////////////////////////////


			//central moments to cumulants
			//4.
			real CUMcbb = mfcbb - ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) / rho;
			real CUMbcb = mfbcb - ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) / rho;
			real CUMbbc = mfbbc - ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) / rho;

			real CUMcca = mfcca - (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho - c1o9*(drho / rho));
			real CUMcac = mfcac - (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho - c1o9*(drho / rho));
			real CUMacc = mfacc - (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho - c1o9*(drho / rho));

			//5.
			real CUMbcc = mfbcc - ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) / rho;
			real CUMcbc = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) / rho;
			real CUMccb = mfccb - ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) / rho;

			//6.

			real CUMccc = mfccc + ((-c4o1 *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
				+ (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
					+ c2o1 * (mfcaa * mfaca * mfaac)
					+ c16o1 *  mfbba * mfbab * mfabb) / (rho * rho)
				- c1o3 * (mfacc + mfcac + mfcca) / rho
				- c1o9 * (mfcaa + mfaca + mfaac) / rho
				+ (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
					+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3
				+ c1o27*((drho * drho - drho) / (rho*rho)));




			//2.
			// linear combinations
			real mxxPyyPzz = mfcaa + mfaca + mfaac;
			real mxxMyy = mfcaa - mfaca;
			real mxxMzz = mfcaa - mfaac;

			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			//incl. correction		(hat noch nicht so gut funktioniert...Optimierungsbedarf??)
			{
				real dxux = c1o2 * (-omega) *(mxxMyy + mxxMzz) + c1o2 *  OxxPyyPzz * (mfaaa - mxxPyyPzz);
				real dyuy = dxux + omega * c3o2 * mxxMyy;
				real dzuz = dxux + omega * c3o2 * mxxMzz;

				//relax
				mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz) - c3o1 * (c1o1 - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2 * dzuz);//-magicBulk*OxxPyyPzz;
				mxxMyy += omega * (-mxxMyy) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy);
				mxxMzz += omega * (-mxxMzz) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz);

			}
			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			////no correction
			//mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz);//-magicBulk*OxxPyyPzz;
			//mxxMyy += -(-omega) * (-mxxMyy);
			//mxxMzz += -(-omega) * (-mxxMzz);
			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			mfabb += omega * (-mfabb);
			mfbab += omega * (-mfbab);
			mfbba += omega * (-mfbba);

			//////////////////////////////////////////////////////////////////////////

			// linear combinations back
			mfcaa = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
			mfaca = c1o3 * (-c2o1*  mxxMyy + mxxMzz + mxxPyyPzz);
			mfaac = c1o3 * (mxxMyy - c2o1* mxxMzz + mxxPyyPzz);

			//3.
			// linear combinations

			real mxxyPyzz = mfcba + mfabc;
			real mxxyMyzz = mfcba - mfabc;

			real mxxzPyyz = mfcab + mfacb;
			real mxxzMyyz = mfcab - mfacb;

			real mxyyPxzz = mfbca + mfbac;
			real mxyyMxzz = mfbca - mfbac;

			//relax
			//////////////////////////////////////////////////////////////////////////
			mfbbb += OxyyMxzz * (-mfbbb);
			mxxyPyzz += OxyyPxzz * (-mxxyPyzz);
			mxxyMyzz += OxyyMxzz * (-mxxyMyzz);
			mxxzPyyz += OxyyPxzz * (-mxxzPyyz);
			mxxzMyyz += OxyyMxzz * (-mxxzMyyz);
			mxyyPxzz += OxyyPxzz * (-mxyyPxzz);
			mxyyMxzz += OxyyMxzz * (-mxyyMxzz);
			//////////////////////////////////////////////////////////////////////////

			mfcba = (mxxyMyzz + mxxyPyzz) * c1o2;
			mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
			mfcab = (mxxzMyyz + mxxzPyyz) * c1o2;
			mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
			mfbca = (mxyyMxzz + mxyyPxzz) * c1o2;
			mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;

			//4.
			//////////////////////////////////////////////////////////////////////////
			CUMacc += O4 * (-CUMacc);
			CUMcac += O4 * (-CUMcac);
			CUMcca += O4 * (-CUMcca);

			CUMbbc += O4 * (-CUMbbc);
			CUMbcb += O4 * (-CUMbcb);
			CUMcbb += O4 * (-CUMcbb);
			//////////////////////////////////////////////////////////////////////////


			//5.
			CUMbcc += O5 * (-CUMbcc);
			CUMcbc += O5 * (-CUMcbc);
			CUMccb += O5 * (-CUMccb);

			//6.
			CUMccc += O6 * (-CUMccc);



			//back cumulants to central moments
			//4.
			mfcbb = CUMcbb + ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) / rho;
			mfbcb = CUMbcb + ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) / rho;
			mfbbc = CUMbbc + ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) / rho;

			mfcca = CUMcca + (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho - c1o9*(drho / rho));
			mfcac = CUMcac + (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho - c1o9*(drho / rho));
			mfacc = CUMacc + (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho - c1o9*(drho / rho));

			//5.
			mfbcc = CUMbcc + ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) / rho;
			mfcbc = CUMcbc + ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) / rho;
			mfccb = CUMccb + ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) / rho;

			//6.
			mfccc = CUMccc - ((-c4o1 *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
				+ (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
					+ c2o1 * (mfcaa * mfaca * mfaac)
					+ c16o1 *  mfbba * mfbab * mfabb) / (rho * rho)
				- c1o3 * (mfacc + mfcac + mfcca) / rho
				- c1o9 * (mfcaa + mfaca + mfaac) / rho
				+ (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
					+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3
				+ c1o27*((drho * drho - drho) / (rho*rho)));

			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			mfbaa = -mfbaa;
			mfaba = -mfaba;
			mfaab = -mfaab;
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			//back
			////////////////////////////////////////////////////////////////////////////////////
			backwardChimera(mfaaa, mfaab, mfaac, vvz, vz2);
			backwardChimera(mfaba, mfabb, mfabc, vvz, vz2);
			backwardChimera(mfaca, mfacb, mfacc, vvz, vz2);
			backwardChimera(mfbaa, mfbab, mfbac, vvz, vz2);
			backwardChimera(mfbba, mfbbb, mfbbc, vvz, vz2);
			backwardChimera(mfbca, mfbcb, mfbcc, vvz, vz2);
			backwardChimera(mfcaa, mfcab, mfcac, vvz, vz2);
			backwardChimera(mfcba, mfcbb, mfcbc, vvz, vz2);
			backwardChimera(mfcca, mfccb, mfccc, vvz, vz2);

			backwardChimera(mfaaa, mfaba, mfaca, vvy, vy2);
			backwardChimera(mfaab, mfabb, mfacb, vvy, vy2);
			backwardChimera(mfaac, mfabc, mfacc, vvy, vy2);
			backwardChimera(mfbaa, mfbba, mfbca, vvy, vy2);
			backwardChimera(mfbab, mfbbb, mfbcb, vvy, vy2);
			backwardChimera(mfbac, mfbbc, mfbcc, vvy, vy2);
			backwardChimera(mfcaa, mfcba, mfcca, vvy, vy2);
			backwardChimera(mfcab, mfcbb, mfccb, vvy, vy2);
			backwardChimera(mfcac, mfcbc, mfccc, vvy, vy2);

			backwardChimera(mfaaa, mfbaa, mfcaa, vvx, vx2);
			backwardChimera(mfaab, mfbab, mfcab, vvx, vx2);
			backwardChimera(mfaac, mfbac, mfcac, vvx, vx2);
			backwardChimera(mfaba, mfbba, mfcba, vvx, vx2);
			backwardChimera(mfabb, mfbbb, mfcbb, vvx, vx2);
			backwardChimera(mfabc, mfbbc, mfcbc, vvx, vx2);
			backwardChimera(mfaca, mfbca, mfcca, vvx, vx2);
			backwardChimera(mfacb, mfbcb, mfccb, vvx, vx2);
			backwardChimera(mfacc, mfbcc, mfccc, vvx, vx2);

			////////////////////////////////////////////////////////////////////////////////////
			mfcbb+=EQcbb;
			mfabb+=EQabb;
			mfbcb+=EQbcb;
			mfbab+=EQbab;
			mfbbc+=EQbbc;
			mfbba+=EQbba;
			mfccb+=EQccb;
			mfaab+=EQaab;
			mfcab+=EQcab;
			mfacb+=EQacb;
			mfcbc+=EQcbc;
			mfaba+=EQaba;
			mfcba+=EQcba;
			mfabc+=EQabc;
			mfbcc+=EQbcc;
			mfbaa+=EQbaa;
			mfbca+=EQbca;
			mfbac+=EQbac;
			mfbbb+=EQbbb;
			mfccc+=EQccc;
			mfaac+=EQaac;
			mfcac+=EQcac;
			mfacc+=EQacc;
			mfcca+=EQcca;
			mfaaa+=EQaaa;
			mfcaa+=EQcaa;
			mfaca+=EQaca;


			//////////////////////////////////////////////////////////////////////////////////////
			////back
			//////////////////////////////////////////////////////////////////////////////////////
			////mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// Z - Dir
			//backwardChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, one);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfaba, mfabb, mfabc, vvz, vz2);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, c1o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbaa, mfbab, mfbac, vvz, vz2);
			///////////b//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbba, mfbbb, mfbbc, vvz, vz2);
			///////////b//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbca, mfbcb, mfbcc, vvz, vz2);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, c1o3);
			///////////c//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfcba, mfcbb, mfcbc, vvz, vz2);
			///////////c//////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			////mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// Y - Dir
			//backwardChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2, c1o6);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaab, mfabb, mfacb, vvy, vy2, c2o3);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, c1o6);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbaa, mfbba, mfbca, vvz, vz2);
			///////////b//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbab, mfbbb, mfbcb, vvz, vz2);
			///////////b//////////////////////////////////////////////////////////////////////////
			//backwardChimera(mfbac, mfbbc, mfbcc, vvz, vz2);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2, c1o18);
			///////////c//////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcab, mfcbb, mfccb, vvy, vy2, c2o9);
			///////////c//////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, c1o18);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			////mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
			//////////////////////////////////////////////////////////////////////////////////////
			//// X - Dir
			//backwardChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaba, mfbba, mfcba, vvx, vx2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaab, mfbab, mfcab, vvx, vx2, c1o9);
			/////////////b////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfabb, mfbbb, mfcbb, vvx, vx2, c4o9);
			/////////////b////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfacb, mfbcb, mfccb, vvx, vx2, c1o9);
			//////////////////////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, c1o36);
			/////////////c////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfabc, mfbbc, mfcbc, vvx, vx2, c1o9);
			/////////////c////////////////////////////////////////////////////////////////////////
			//backwardChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, c1o36);
			//////////////////////////////////////////////////////////////////////////////////////

			//////////////////////////////////////////////////////////////////////////////////////
			real drhoPost =
				((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
					((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;
			mfbbb += drho - drhoPost;
			////////////////////////////////////////////////////////////////////////////////////
			(D.f[dirE])[k] = mfabb;//(D.f[ dirE   ])[ke   ] = mfabb;// -  c2over27 ;  (D.f[ dirE   ])[k   ]                                                                     
			(D.f[dirW])[kw] = mfcbb;//(D.f[ dirW   ])[kw   ] = mfcbb;// -  c2over27 ;  (D.f[ dirW   ])[kw  ]                                                                   
			(D.f[dirN])[k] = mfbab;//(D.f[ dirN   ])[kn   ] = mfbab;// -  c2over27 ;	 (D.f[ dirN   ])[k   ]
			(D.f[dirS])[ks] = mfbcb;//(D.f[ dirS   ])[ks   ] = mfbcb;// -  c2over27 ;	 (D.f[ dirS   ])[ks  ]
			(D.f[dirT])[k] = mfbba;//(D.f[ dirT   ])[kt   ] = mfbba;// -  c2over27 ;	 (D.f[ dirT   ])[k   ]
			(D.f[dirB])[kb] = mfbbc;//(D.f[ dirB   ])[kb   ] = mfbbc;// -  c2over27 ;	 (D.f[ dirB   ])[kb  ]
			(D.f[dirNE])[k] = mfaab;//(D.f[ dirNE  ])[kne  ] = mfaab;// -  c1over54 ;	 (D.f[ dirNE  ])[k   ]
			(D.f[dirSW])[ksw] = mfccb;//(D.f[ dirSW  ])[ksw  ] = mfccb;// -  c1over54 ;	 (D.f[ dirSW  ])[ksw ]
			(D.f[dirSE])[ks] = mfacb;//(D.f[ dirSE  ])[kse  ] = mfacb;// -  c1over54 ;	 (D.f[ dirSE  ])[ks  ]
			(D.f[dirNW])[kw] = mfcab;//(D.f[ dirNW  ])[knw  ] = mfcab;// -  c1over54 ;	 (D.f[ dirNW  ])[kw  ]
			(D.f[dirTE])[k] = mfaba;//(D.f[ dirTE  ])[kte  ] = mfaba;// -  c1over54 ;	 (D.f[ dirTE  ])[k   ]
			(D.f[dirBW])[kbw] = mfcbc;//(D.f[ dirBW  ])[kbw  ] = mfcbc;// -  c1over54 ;	 (D.f[ dirBW  ])[kbw ]
			(D.f[dirBE])[kb] = mfabc;//(D.f[ dirBE  ])[kbe  ] = mfabc;// -  c1over54 ;	 (D.f[ dirBE  ])[kb  ]
			(D.f[dirTW])[kw] = mfcba;//(D.f[ dirTW  ])[ktw  ] = mfcba;// -  c1over54 ;	 (D.f[ dirTW  ])[kw  ]
			(D.f[dirTN])[k] = mfbaa;//(D.f[ dirTN  ])[ktn  ] = mfbaa;// -  c1over54 ;	 (D.f[ dirTN  ])[k   ]
			(D.f[dirBS])[kbs] = mfbcc;//(D.f[ dirBS  ])[kbs  ] = mfbcc;// -  c1over54 ;	 (D.f[ dirBS  ])[kbs ]
			(D.f[dirBN])[kb] = mfbac;//(D.f[ dirBN  ])[kbn  ] = mfbac;// -  c1over54 ;	 (D.f[ dirBN  ])[kb  ]
			(D.f[dirTS])[ks] = mfbca;//(D.f[ dirTS  ])[kts  ] = mfbca;// -  c1over54 ;	 (D.f[ dirTS  ])[ks  ]
			(D.f[dirZERO])[k] = mfbbb;//(D.f[ dirZERO])[kzero] = mfbbb;// -  c8over27 ;	 (D.f[ dirZERO])[k   ]
			(D.f[dirTNE])[k] = mfaaa;//(D.f[ dirTNE ])[ktne ] = mfaaa;// -  c1over216;	 (D.f[ dirTNE ])[k   ]
			(D.f[dirTSE])[ks] = mfaca;//(D.f[ dirTSE ])[ktse ] = mfaca;// -  c1over216;	 (D.f[ dirTSE ])[ks  ]
			(D.f[dirBNE])[kb] = mfaac;//(D.f[ dirBNE ])[kbne ] = mfaac;// -  c1over216;	 (D.f[ dirBNE ])[kb  ]
			(D.f[dirBSE])[kbs] = mfacc;//(D.f[ dirBSE ])[kbse ] = mfacc;// -  c1over216;	 (D.f[ dirBSE ])[kbs ]
			(D.f[dirTNW])[kw] = mfcaa;//(D.f[ dirTNW ])[ktnw ] = mfcaa;// -  c1over216;	 (D.f[ dirTNW ])[kw  ]
			(D.f[dirTSW])[ksw] = mfcca;//(D.f[ dirTSW ])[ktsw ] = mfcca;// -  c1over216;	 (D.f[ dirTSW ])[ksw ]
			(D.f[dirBNW])[kbw] = mfcac;//(D.f[ dirBNW ])[kbnw ] = mfcac;// -  c1over216;	 (D.f[ dirBNW ])[kbw ]
			(D.f[dirBSW])[kbsw] = mfccc;//(D.f[ dirBSW ])[kbsw ] = mfccc;// -  c1over216;	 (D.f[ dirBSW ])[kbsw]
			////////////////////////////////////////////////////////////////////////////////////
		}
	}
}
////////////////////////////////////////////////////////////////////////////////








































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void Cumulant_One_chim_Comp_SP_27(
	real omega,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* DDStart,
	int size_Mat,
	int level,
	real* forces,
	bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k<size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if (BC >= GEO_FLUID/*(BC != GEO_SOLID) && (BC != GEO_VOID)*/)
		{
			Distributions27 D;
			if (EvenOrOdd == true)
			{
				D.f[dirE   ] = &DDStart[dirE   *size_Mat];
				D.f[dirW   ] = &DDStart[dirW   *size_Mat];
				D.f[dirN   ] = &DDStart[dirN   *size_Mat];
				D.f[dirS   ] = &DDStart[dirS   *size_Mat];
				D.f[dirT   ] = &DDStart[dirT   *size_Mat];
				D.f[dirB   ] = &DDStart[dirB   *size_Mat];
				D.f[dirNE  ] = &DDStart[dirNE  *size_Mat];
				D.f[dirSW  ] = &DDStart[dirSW  *size_Mat];
				D.f[dirSE  ] = &DDStart[dirSE  *size_Mat];
				D.f[dirNW  ] = &DDStart[dirNW  *size_Mat];
				D.f[dirTE  ] = &DDStart[dirTE  *size_Mat];
				D.f[dirBW  ] = &DDStart[dirBW  *size_Mat];
				D.f[dirBE  ] = &DDStart[dirBE  *size_Mat];
				D.f[dirTW  ] = &DDStart[dirTW  *size_Mat];
				D.f[dirTN  ] = &DDStart[dirTN  *size_Mat];
				D.f[dirBS  ] = &DDStart[dirBS  *size_Mat];
				D.f[dirBN  ] = &DDStart[dirBN  *size_Mat];
				D.f[dirTS  ] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirTNE ] = &DDStart[dirTNE *size_Mat];
				D.f[dirTSW ] = &DDStart[dirTSW *size_Mat];
				D.f[dirTSE ] = &DDStart[dirTSE *size_Mat];
				D.f[dirTNW ] = &DDStart[dirTNW *size_Mat];
				D.f[dirBNE ] = &DDStart[dirBNE *size_Mat];
				D.f[dirBSW ] = &DDStart[dirBSW *size_Mat];
				D.f[dirBSE ] = &DDStart[dirBSE *size_Mat];
				D.f[dirBNW ] = &DDStart[dirBNW *size_Mat];
			}
			else
			{
				D.f[dirW   ] = &DDStart[dirE   *size_Mat];
				D.f[dirE   ] = &DDStart[dirW   *size_Mat];
				D.f[dirS   ] = &DDStart[dirN   *size_Mat];
				D.f[dirN   ] = &DDStart[dirS   *size_Mat];
				D.f[dirB   ] = &DDStart[dirT   *size_Mat];
				D.f[dirT   ] = &DDStart[dirB   *size_Mat];
				D.f[dirSW  ] = &DDStart[dirNE  *size_Mat];
				D.f[dirNE  ] = &DDStart[dirSW  *size_Mat];
				D.f[dirNW  ] = &DDStart[dirSE  *size_Mat];
				D.f[dirSE  ] = &DDStart[dirNW  *size_Mat];
				D.f[dirBW  ] = &DDStart[dirTE  *size_Mat];
				D.f[dirTE  ] = &DDStart[dirBW  *size_Mat];
				D.f[dirTW  ] = &DDStart[dirBE  *size_Mat];
				D.f[dirBE  ] = &DDStart[dirTW  *size_Mat];
				D.f[dirBS  ] = &DDStart[dirTN  *size_Mat];
				D.f[dirTN  ] = &DDStart[dirBS  *size_Mat];
				D.f[dirTS  ] = &DDStart[dirBN  *size_Mat];
				D.f[dirBN  ] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirBSW ] = &DDStart[dirTNE *size_Mat];
				D.f[dirBNE ] = &DDStart[dirTSW *size_Mat];
				D.f[dirBNW ] = &DDStart[dirTSE *size_Mat];
				D.f[dirBSE ] = &DDStart[dirTNW *size_Mat];
				D.f[dirTSW ] = &DDStart[dirBNE *size_Mat];
				D.f[dirTNE ] = &DDStart[dirBSW *size_Mat];
				D.f[dirTNW ] = &DDStart[dirBSE *size_Mat];
				D.f[dirTSE ] = &DDStart[dirBNW *size_Mat];
			}
			////////////////////////////////////////////////////////////////////////////////
			//index
			unsigned int kw = neighborX[k];
			unsigned int ks = neighborY[k];
			unsigned int kb = neighborZ[k];
			unsigned int ksw = neighborY[kw];
			unsigned int kbw = neighborZ[kw];
			unsigned int kbs = neighborZ[ks];
			unsigned int kbsw = neighborZ[ksw];
			////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[dirE   ])[k   ];
			real mfabb = (D.f[dirW   ])[kw  ];
			real mfbcb = (D.f[dirN   ])[k   ];
			real mfbab = (D.f[dirS   ])[ks  ];
			real mfbbc = (D.f[dirT   ])[k   ];
			real mfbba = (D.f[dirB   ])[kb  ];
			real mfccb = (D.f[dirNE  ])[k   ];
			real mfaab = (D.f[dirSW  ])[ksw ];
			real mfcab = (D.f[dirSE  ])[ks  ];
			real mfacb = (D.f[dirNW  ])[kw  ];
			real mfcbc = (D.f[dirTE  ])[k   ];
			real mfaba = (D.f[dirBW  ])[kbw ];
			real mfcba = (D.f[dirBE  ])[kb  ];
			real mfabc = (D.f[dirTW  ])[kw  ];
			real mfbcc = (D.f[dirTN  ])[k   ];
			real mfbaa = (D.f[dirBS  ])[kbs ];
			real mfbca = (D.f[dirBN  ])[kb  ];
			real mfbac = (D.f[dirTS  ])[ks  ];
			real mfbbb = (D.f[dirZERO])[k   ];
			real mfccc = (D.f[dirTNE ])[k   ];
			real mfaac = (D.f[dirTSW ])[ksw ];
			real mfcac = (D.f[dirTSE ])[ks  ];
			real mfacc = (D.f[dirTNW ])[kw  ];
			real mfcca = (D.f[dirBNE ])[kb  ];
			real mfaaa = (D.f[dirBSW ])[kbsw];
			real mfcaa = (D.f[dirBSE ])[kbs ];
			real mfaca = (D.f[dirBNW ])[kbw ];
			////////////////////////////////////////////////////////////////////////////////////
			real drho = ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;

			real rho = c1o1 + drho;
			real OOrho = c1o1 / rho;
			////////////////////////////////////////////////////////////////////////////////////
			real vvx = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
				(((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
				(mfcbb - mfabb)) * OOrho;
			real vvy = ((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
				(((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
				(mfbcb - mfbab)) * OOrho;
			real vvz = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
				(((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
				(mfbbc - mfbba)) * OOrho;
			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			real fx = forces[0] / (pow((double)c2o1, (double)level)); //zero;//0.0032653/(pow(two,level)); //0.000000005;//(two/1600000.0) / 120.0; //
			real fy = forces[1] / (pow((double)c2o1, (double)level)); //zero;
			real fz = forces[2] / (pow((double)c2o1, (double)level)); //zero;
			vvx += fx*c1o2;
			vvy += fy*c1o2;
			vvz += fz*c1o2;
			////////////////////////////////////////////////////////////////////////////////////
			//real omega = omega_in;
			////////////////////////////////////////////////////////////////////////////////////
			//fast
			real oMdrho = c1o1; // comp special
			real m0, m1, m2;
			real vx2;
			real vy2;
			real vz2;
			vx2 = vvx*vvx;
			vy2 = vvy*vvy;
			vz2 = vvz*vvz;
			////////////////////////////////////////////////////////////////////////////////////
			real wadjust;
			real qudricLimitP = c1o100;// * 0.0001f;
			real qudricLimitM = c1o100;// * 0.0001f;
			real qudricLimitD = c1o100;// * 0.001f;
			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			forwardInverseChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, 36.0f, c1o36);
			forwardInverseChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2, 9.0f , c1o9 );
			forwardInverseChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, 36.0f, c1o36);
			forwardInverseChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2, 9.0f , c1o9 );
			forwardInverseChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2, 2.25f, c4o9 );
			forwardInverseChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2, 9.0f , c1o9 );
			forwardInverseChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, 36.0f, c1o36);
			forwardInverseChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2, 9.0f , c1o9 );
			forwardInverseChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, 36.0f, c1o36);

			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			forwardInverseChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2, 6.0f , c1o6 );
			forwardChimera(     mfaab, mfabb, mfacb, vvy, vy2);
			forwardInverseChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, 18.0f, c1o18);
			forwardInverseChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2, 1.5f , c2o3 );
			forwardChimera(     mfbab, mfbbb, mfbcb, vvy, vy2);
			forwardInverseChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2, 4.5f , c2o9 );
			forwardInverseChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2, 6.0f , c1o6 );
			forwardChimera(     mfcab, mfcbb, mfccb, vvy, vy2);
			forwardInverseChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, 18.0f, c1o18);

			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			forwardInverseChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o1, c1o1);
			forwardChimera(     mfaba, mfbba, mfcba, vvx, vx2);
			forwardInverseChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, 3.0f, c1o3);
			forwardChimera(     mfaab, mfbab, mfcab, vvx, vx2);
			forwardChimera(     mfabb, mfbbb, mfcbb, vvx, vx2);
			forwardChimera(     mfacb, mfbcb, mfccb, vvx, vx2);
			forwardInverseChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, 3.0f, c1o3);
			forwardChimera(     mfabc, mfbbc, mfcbc, vvx, vx2);
			forwardInverseChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, 9.0f, c1o9);

			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			real OxxPyyPzz = c1o1;
			////////////////////////////////////////////////////////////
			//3.
			//////////////////////////////
			real OxyyPxzz = c1o1;
			real OxyyMxzz = c1o1;
			real Oxyz = c1o1;
			////////////////////////////////////////////////////////////
			//4.
			//////////////////////////////
			real O4 = c1o1;
			////////////////////////////////////////////////////////////
			//5.
			//////////////////////////////
			real O5 = c1o1;
			////////////////////////////////////////////////////////////
			//6.
			//////////////////////////////
			real O6 = c1o1;
			////////////////////////////////////////////////////////////


			//central moments to cumulants
			//4.
			real CUMcbb = mfcbb - ((mfcaa + c1o3) * mfabb + c2o1 * mfbba * mfbab) * OOrho;
			real CUMbcb = mfbcb - ((mfaca + c1o3) * mfbab + c2o1 * mfbba * mfabb) * OOrho;
			real CUMbbc = mfbbc - ((mfaac + c1o3) * mfbba + c2o1 * mfbab * mfabb) * OOrho;

			real CUMcca = mfcca - (((mfcaa * mfaca + c2o1 * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) * OOrho - c1o9*(drho * OOrho));
			real CUMcac = mfcac - (((mfcaa * mfaac + c2o1 * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) * OOrho - c1o9*(drho * OOrho));
			real CUMacc = mfacc - (((mfaac * mfaca + c2o1 * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) * OOrho - c1o9*(drho * OOrho));

			//5.
			real CUMbcc = mfbcc - ((mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) * OOrho;
			real CUMcbc = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) * OOrho;
			real CUMccb = mfccb - ((mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) * OOrho;

			//6.
			real CUMccc = mfccc + ((-c4o1 *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) * OOrho
				+ (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
				+ c2o1 * (mfcaa * mfaca * mfaac)
				+ c16o1 *  mfbba * mfbab * mfabb) * OOrho * OOrho
				- c1o3 * (mfacc + mfcac + mfcca) * OOrho
				- c1o9 * (mfcaa + mfaca + mfaac) * OOrho
				+ (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
				+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) * OOrho * OOrho  * c2o3
				+ c1o27*((drho * drho - drho) * OOrho * OOrho ));


			//2.
			// linear combinations
			real mxxPyyPzz = mfcaa + mfaca + mfaac;
			real mxxMyy = mfcaa - mfaca;
			real mxxMzz = mfcaa - mfaac;

			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			//incl. correction		(hat noch nicht so gut funktioniert...Optimierungsbedarf??)
			{
				real dxux = c1o2 * (-omega) *(mxxMyy + mxxMzz) + c1o2 *  OxxPyyPzz * (mfaaa - mxxPyyPzz);
				real dyuy = dxux + omega * c3o2 * mxxMyy;
				real dzuz = dxux + omega * c3o2 * mxxMzz;

				//relax
				mxxPyyPzz += OxxPyyPzz*(mfaaa  - mxxPyyPzz)- c3o1 * (c1o1 - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2 * dzuz);//-magicBulk*OxxPyyPzz;
				mxxMyy    += omega * (-mxxMyy) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy);
				mxxMzz    += omega * (-mxxMzz) - c3o1 * (c1o1 + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz);

			}
			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			////no correction
			//mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz);//-magicBulk*OxxPyyPzz;
			//mxxMyy += -(-omega) * (-mxxMyy);
			//mxxMzz += -(-omega) * (-mxxMzz);
			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			mfabb += omega * (-mfabb);
			mfbab += omega * (-mfbab);
			mfbba += omega * (-mfbba);

			//////////////////////////////////////////////////////////////////////////

			// linear combinations back
			mfcaa = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
			mfaca = c1o3 * (-c2o1*  mxxMyy + mxxMzz + mxxPyyPzz);
			mfaac = c1o3 * (mxxMyy - c2o1* mxxMzz + mxxPyyPzz);

			//3.
			// linear combinations

			real mxxyPyzz = mfcba + mfabc;
			real mxxyMyzz = mfcba - mfabc;

			real mxxzPyyz = mfcab + mfacb;
			real mxxzMyyz = mfcab - mfacb;

			real mxyyPxzz = mfbca + mfbac;
			real mxyyMxzz = mfbca - mfbac;

			//relax
			//////////////////////////////////////////////////////////////////////////
			mfbbb     += OxyyMxzz * (-mfbbb);
			mxxyPyzz  += OxyyPxzz * (-mxxyPyzz);
			mxxyMyzz  += OxyyMxzz * (-mxxyMyzz);
			mxxzPyyz  += OxyyPxzz * (-mxxzPyyz);
			mxxzMyyz  += OxyyMxzz * (-mxxzMyyz);
			mxyyPxzz  += OxyyPxzz * (-mxyyPxzz);
			mxyyMxzz  += OxyyMxzz * (-mxyyMxzz);
			//////////////////////////////////////////////////////////////////////////

			mfcba = (mxxyMyzz + mxxyPyzz) * c1o2;
			mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
			mfcab = (mxxzMyyz + mxxzPyyz) * c1o2;
			mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
			mfbca = (mxyyMxzz + mxyyPxzz) * c1o2;
			mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;

			//4.
			//////////////////////////////////////////////////////////////////////////
			CUMacc += O4 * (-CUMacc);
			CUMcac += O4 * (-CUMcac);
			CUMcca += O4 * (-CUMcca);

			CUMbbc += O4 * (-CUMbbc);
			CUMbcb += O4 * (-CUMbcb);
			CUMcbb += O4 * (-CUMcbb);
			//////////////////////////////////////////////////////////////////////////


			//5.
			CUMbcc += O5 * (-CUMbcc);
			CUMcbc += O5 * (-CUMcbc);
			CUMccb += O5 * (-CUMccb);

			//6.
			CUMccc += O6 * (-CUMccc);



			//back cumulants to central moments
			//4.
			mfcbb = CUMcbb + c1o3*((c3o1*mfcaa + c1o1) * mfabb + c6o1 * mfbba * mfbab) * OOrho; 
			mfbcb = CUMbcb + c1o3*((c3o1*mfaca + c1o1) * mfbab + c6o1 * mfbba * mfabb) * OOrho;
			mfbbc = CUMbbc + c1o3*((c3o1*mfaac + c1o1) * mfbba + c6o1 * mfbab * mfabb) * OOrho;

			mfcca = CUMcca + (((mfcaa * mfaca + c2o1 * mfbba * mfbba)*c9o1 + c3o1 * (mfcaa + mfaca)) * OOrho - (drho * OOrho))*c1o9;
			mfcac = CUMcac + (((mfcaa * mfaac + c2o1 * mfbab * mfbab)*c9o1 + c3o1 * (mfcaa + mfaac)) * OOrho - (drho * OOrho))*c1o9;
			mfacc = CUMacc + (((mfaac * mfaca + c2o1 * mfabb * mfabb)*c9o1 + c3o1 * (mfaac + mfaca)) * OOrho - (drho * OOrho))*c1o9;

			//5.
			mfbcc = CUMbcc + c1o3 *(c3o1*(mfaac * mfbca + mfaca * mfbac + c4o1 * mfabb * mfbbb + c2o1 * (mfbab * mfacb + mfbba * mfabc)) + (mfbca + mfbac)) * OOrho;
			mfcbc = CUMcbc + c1o3 *(c3o1*(mfaac * mfcba + mfcaa * mfabc + c4o1 * mfbab * mfbbb + c2o1 * (mfabb * mfcab + mfbba * mfbac)) + (mfcba + mfabc)) * OOrho;
			mfccb = CUMccb + c1o3 *(c3o1*(mfcaa * mfacb + mfaca * mfcab + c4o1 * mfbba * mfbbb + c2o1 * (mfbab * mfbca + mfabb * mfcba)) +  (mfacb + mfcab)) * OOrho;

			//6.
			mfccc = 
				CUMccc - ((-c4o1 *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- c4o1 * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- c2o1 * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) * OOrho
				+ (c4o1 * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
				+ c2o1 * (mfcaa * mfaca * mfaac)
				+ c16o1 *  mfbba * mfbab * mfabb) * OOrho * OOrho
				- c1o3 * (mfacc + mfcac + mfcca) * OOrho
				- c1o9 * (mfcaa + mfaca + mfaac) * OOrho
				+ (c2o1 * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
				+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) * OOrho * OOrho * c2o3
				+ c1o27*((drho * drho - drho) * OOrho * OOrho ));

			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			mfbaa = -mfbaa;
			mfaba = -mfaba;
			mfaab = -mfaab;
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			//back
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			backwardInverseChimeraWithK(mfaaa, mfbaa, mfcaa, vvx, vx2, c1o1, c1o1);
			backwardChimera(			mfaba, mfbba, mfcba, vvx, vx2);
			backwardInverseChimeraWithK(mfaca, mfbca, mfcca, vvx, vx2, 3.0f, c1o3);
			backwardChimera(			mfaab, mfbab, mfcab, vvx, vx2);
			backwardChimera(			mfabb, mfbbb, mfcbb, vvx, vx2);
			backwardChimera(			mfacb, mfbcb, mfccb, vvx, vx2);
			backwardInverseChimeraWithK(mfaac, mfbac, mfcac, vvx, vx2, 3.0f, c1o3);
			backwardChimera(			mfabc, mfbbc, mfcbc, vvx, vx2);
			backwardInverseChimeraWithK(mfacc, mfbcc, mfccc, vvx, vx2, 9.0f, c1o9);

			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			backwardInverseChimeraWithK(mfaaa, mfaba, mfaca, vvy, vy2, 6.0f , c1o6 );
			backwardChimera(			mfaab, mfabb, mfacb, vvy, vy2);
			backwardInverseChimeraWithK(mfaac, mfabc, mfacc, vvy, vy2, 18.0f, c1o18);
			backwardInverseChimeraWithK(mfbaa, mfbba, mfbca, vvy, vy2, 1.5f , c2o3 );
			backwardChimera(			mfbab, mfbbb, mfbcb, vvy, vy2);
			backwardInverseChimeraWithK(mfbac, mfbbc, mfbcc, vvy, vy2, 4.5f , c2o9 );
			backwardInverseChimeraWithK(mfcaa, mfcba, mfcca, vvy, vy2, 6.0f , c1o6 );
			backwardChimera(			mfcab, mfcbb, mfccb, vvy, vy2);
			backwardInverseChimeraWithK(mfcac, mfcbc, mfccc, vvy, vy2, 18.0f, c1o18);

			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			backwardInverseChimeraWithK(mfaaa, mfaab, mfaac, vvz, vz2, 36.0f, c1o36);
			backwardInverseChimeraWithK(mfaba, mfabb, mfabc, vvz, vz2, 9.0f , c1o9 );
			backwardInverseChimeraWithK(mfaca, mfacb, mfacc, vvz, vz2, 36.0f, c1o36);
			backwardInverseChimeraWithK(mfbaa, mfbab, mfbac, vvz, vz2, 9.0f , c1o9 );
			backwardInverseChimeraWithK(mfbba, mfbbb, mfbbc, vvz, vz2, 2.25f, c4o9 );
			backwardInverseChimeraWithK(mfbca, mfbcb, mfbcc, vvz, vz2, 9.0f , c1o9 );
			backwardInverseChimeraWithK(mfcaa, mfcab, mfcac, vvz, vz2, 36.0f, c1o36);
			backwardInverseChimeraWithK(mfcba, mfcbb, mfcbc, vvz, vz2, 9.0f , c1o9 );
			backwardInverseChimeraWithK(mfcca, mfccb, mfccc, vvz, vz2, 36.0f, c1o36);

			//////////////////////////////////////////////////////////////////////////////////////
			real drhoPost =
				((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
					((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;
			mfbbb += drho - drhoPost;
			////////////////////////////////////////////////////////////////////////////////////
			(D.f[dirE   ])[k   ] = mfabb;                                                                   
			(D.f[dirW   ])[kw  ] = mfcbb;                                                                 
			(D.f[dirN   ])[k   ] = mfbab;
			(D.f[dirS   ])[ks  ] = mfbcb;
			(D.f[dirT   ])[k   ] = mfbba;
			(D.f[dirB   ])[kb  ] = mfbbc;
			(D.f[dirNE  ])[k   ] = mfaab;
			(D.f[dirSW  ])[ksw ] = mfccb;
			(D.f[dirSE  ])[ks  ] = mfacb;
			(D.f[dirNW  ])[kw  ] = mfcab;
			(D.f[dirTE  ])[k   ] = mfaba;
			(D.f[dirBW  ])[kbw ] = mfcbc;
			(D.f[dirBE  ])[kb  ] = mfabc;
			(D.f[dirTW  ])[kw  ] = mfcba;
			(D.f[dirTN  ])[k   ] = mfbaa;
			(D.f[dirBS  ])[kbs ] = mfbcc;
			(D.f[dirBN  ])[kb  ] = mfbac;
			(D.f[dirTS  ])[ks  ] = mfbca;
			(D.f[dirZERO])[k   ] = mfbbb;
			(D.f[dirTNE ])[k   ] = mfaaa;
			(D.f[dirTSE ])[ks  ] = mfaca;
			(D.f[dirBNE ])[kb  ] = mfaac;
			(D.f[dirBSE ])[kbs ] = mfacc;
			(D.f[dirTNW ])[kw  ] = mfcaa;
			(D.f[dirTSW ])[ksw ] = mfcca;
			(D.f[dirBNW ])[kbw ] = mfcac;
			(D.f[dirBSW ])[kbsw] = mfccc;
		}
	}
}
////////////////////////////////////////////////////////////////////////////////








































