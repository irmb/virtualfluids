#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file StressBcs27.cu
//! \author Henrik Asmuth
//! \date 16/05/2022
//! \brief Kernels for StressBC using the iMEM approach
//!
//! Both kernels prescribe a wall shear stress using the iMEM apprach (see, Asmuth et. al (2021), https://doi.org/10.1063/5.0065701)
//! QStressDeviceComp27 couples the iMEM to the single-node interpolated bounce-back.
//! BBStressDevice27 couples the iMEM to a simple bounce-back.
//! Note, that the iMEM function is currently only implemented for straight walls with z-normal and q=0.5.
//! Other wall models could be implemented in the iMEM by replacing the formulations from Monin-Obukhov similarity theory (MOST)
//! with other formulations, e.g., for smooth walls.
//! iMEM so far most extensively tested with BBStressDevice27, but QStressDeviceComp27 also seems to be stable and working.
//=======================================================================================

#include "LBM/LB.h"
#include "lbm/constants/D3Q27.h"
#include "Kernel/Utilities/DistributionHelper.cuh"
#include <lbm/constants/NumericConstants.h>
#include "KernelUtilities.h"

using namespace vf::lbm::constant;
using namespace vf::lbm::dir;

//////////////////////////////////////////////////////////////////////////////
__host__ __device__ __forceinline__ void iMEM(uint k, uint kN,
                                                         real* _wallNormalX, real* _wallNormalY, real* _wallNormalZ,
                                                         real* vx, real* vy, real* vz,
                                                         real* vx_el,      real* vy_el,      real* vz_el,      //!>mean (temporally filtered) velocities at exchange location
                                                         real* vx_w_mean,  real* vy_w_mean,  real* vz_w_mean,  //!>mean (temporally filtered) velocities at wall-adjactent node
                                                         real  vx_w_inst,  real  vy_w_inst,  real  vz_w_inst,  //!>instantaneous velocities at wall-adjactent node
                                                         real  rho,
                                                         int* samplingOffset,
                                                         real q,
                                                         real forceFactor,                                     //!>e.g., 1.0 for simple-bounce back, or (1+q) for interpolated single-node bounce-back as in Geier et al (2015)
                                                         real eps,                                             //!>filter constant in temporal averaging
                                                         real* z0,                                             //!>aerodynamic roughness length
                                                         bool  hasWallModelMonitor,
                                                         real* u_star_monitor,
                                                         real wallMomentumX, real wallMomentumY, real wallMomentumZ,
                                                         real& wallVelocityX, real& wallVelocityY, real&wallVelocityZ)
{
      real wallNormalX = _wallNormalX[k];
      real wallNormalY = _wallNormalY[k];
      real wallNormalZ = _wallNormalZ[k];

      //Sample velocity at exchange location and filter temporally
      real _vx_el = eps*vx[kN]+(1.0-eps)*vx_el[k];
      real _vy_el = eps*vy[kN]+(1.0-eps)*vy_el[k];
      real _vz_el = eps*vz[kN]+(1.0-eps)*vz_el[k];
      vx_el[k] = _vx_el;
      vy_el[k] = _vy_el;
      vz_el[k] = _vz_el;

      //filter velocity at wall-adjacent node
      real _vx_w_mean = eps*vx_w_inst+(1.0-eps)*vx_w_mean[k];
      real _vy_w_mean = eps*vy_w_inst+(1.0-eps)*vy_w_mean[k];
      real _vz_w_mean = eps*vz_w_inst+(1.0-eps)*vz_w_mean[k];
      vx_w_mean[k] = _vx_w_mean;
      vy_w_mean[k] = _vy_w_mean;
      vz_w_mean[k] = _vz_w_mean;

      //Subtract wall-normal velocity components
      real vDotN_el = _vx_el*wallNormalX + _vy_el*wallNormalY + _vz_el*wallNormalZ;
      _vx_el -= vDotN_el*wallNormalX;
      _vy_el -= vDotN_el*wallNormalY;
      _vz_el -= vDotN_el*wallNormalZ;
      real vMag_el = sqrt( _vx_el*_vx_el + _vy_el*_vy_el + _vz_el*_vz_el );

      real vDotN_w_mean = _vx_w_mean*wallNormalX + _vy_w_mean*wallNormalY + _vz_w_mean*wallNormalZ;
      _vx_w_mean -= vDotN_w_mean*wallNormalX;
      _vy_w_mean -= vDotN_w_mean*wallNormalY;
      _vz_w_mean -= vDotN_w_mean*wallNormalZ;
      real vMag_w_mean = sqrt( _vx_w_mean*_vx_w_mean + _vy_w_mean*_vy_w_mean + _vz_w_mean*_vz_w_mean );

      real vDotN_w = vx_w_inst*wallNormalX + vy_w_inst*wallNormalY + vz_w_inst*wallNormalZ;
      real _vx_w = vx_w_inst-vDotN_w*wallNormalX;
      real _vy_w = vy_w_inst-vDotN_w*wallNormalY;
      real _vz_w = vz_w_inst-vDotN_w*wallNormalZ;

      //Compute wall shear stress tau_w via MOST
      real z = (real)samplingOffset[k] + q; //assuming q=0.5, could be replaced by wall distance via wall normal
      real kappa = 0.4;
      real u_star = vMag_el*kappa/(log(z/z0[k]));
      if(hasWallModelMonitor) u_star_monitor[k] = u_star;
      real tau_w = u_star*u_star;                  //Note: this is actually tau_w/rho
      real A = 1.0;                                //wall area (obviously 1 for grid aligned walls, can come from grid builder later for complex geometries)

      //Scale wall shear stress with near wall velocity, i.e., Schumann-Grötzbach (SG) approach
      real F_w_x = (tau_w*A) * (_vx_w/vMag_w_mean);//(_vx_el/vMag_el)
      real F_w_y = (tau_w*A) * (_vy_w/vMag_w_mean);//(_vy_el/vMag_el)
      real F_w_z = (tau_w*A) * (_vz_w/vMag_w_mean);//(_vz_el/vMag_el)
      //                                                ^^^^^^^^^^^^--- old alternative: do not scale SG-like but only set direction via velocity at exchange location

      //Momentum to be applied via wall velocity
      real wallMomDotN = wallMomentumX*wallNormalX+wallMomentumY*wallNormalY+wallMomentumZ*wallNormalZ;
      real F_x =  F_w_x - ( wallMomentumX - wallMomDotN*wallNormalX )/rho;
      real F_y =  F_w_y - ( wallMomentumY - wallMomDotN*wallNormalY )/rho;
      real F_z =  F_w_z - ( wallMomentumZ - wallMomDotN*wallNormalZ )/rho;

      //Compute  wall velocity and clip (clipping only necessary for initial boundary layer development)
      real clipWallVelo = 2.0;
      real clipVx = clipWallVelo*_vx_el;
      real clipVy = clipWallVelo*_vy_el;
      real clipVz = clipWallVelo*_vz_el;

      wallVelocityX = clipVx > -clipVx? min(clipVx, max(-clipVx, -3.0*F_x*forceFactor)): max(clipVx, min(-clipVx, -3.0*F_x*forceFactor));
      wallVelocityY = clipVy > -clipVy? min(clipVy, max(-clipVy, -3.0*F_y*forceFactor)): max(clipVy, min(-clipVy, -3.0*F_y*forceFactor));
      wallVelocityZ = clipVz > -clipVz? min(clipVz, max(-clipVz, -3.0*F_z*forceFactor)): max(clipVz, min(-clipVz, -3.0*F_z*forceFactor));
}


//////////////////////////////////////////////////////////////////////////////
__global__ void QStressDeviceComp27(real* DD,
											   int* k_Q,
                                    int* k_N,
											   real* QQ,
                                    unsigned int numberOfBCnodes,
                                    real om1,
                                    real* turbViscosity,
                                    real* vx,
                                    real* vy,
                                    real* vz,
                                    real* normalX,
                                    real* normalY,
                                    real* normalZ,
                                    real* vx_el,
                                    real* vy_el,
                                    real* vz_el,
                                    real* vx_w_mean,
                                    real* vy_w_mean,
                                    real* vz_w_mean,
                                    int* samplingOffset,
                                    real* z0,
                                    bool  hasWallModelMonitor,
                                    real* u_star_monitor,
                                    real* Fx_monitor,
                                    real* Fy_monitor,
                                    real* Fz_monitor,
											   unsigned int* neighborX,
                                    unsigned int* neighborY,
                                    unsigned int* neighborZ,
                                    unsigned long long numberOfLBnodes,
                                    bool isEvenTimestep)
{

   Distributions27 D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, isEvenTimestep);

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index
   const unsigned  y = blockIdx.x;   // Globaler y-Index
   const unsigned  z = blockIdx.y;   // Globaler z-Index

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes/*numberOfBCnodes*/)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB,
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW;
      q_dirE   = &QQ[DIR_P00 * numberOfBCnodes];
      q_dirW   = &QQ[DIR_M00 * numberOfBCnodes];
      q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;      //get right adress of post-coll f's
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[DIR_P00])[ke   ];     //post-coll f's
      f_E    = (D.f[DIR_M00])[kw   ];
      f_S    = (D.f[DIR_0P0])[kn   ];
      f_N    = (D.f[DIR_0M0])[ks   ];
      f_B    = (D.f[DIR_00P])[kt   ];
      f_T    = (D.f[DIR_00M])[kb   ];
      f_SW   = (D.f[DIR_PP0])[kne  ];
      f_NE   = (D.f[DIR_MM0])[ksw  ];
      f_NW   = (D.f[DIR_PM0])[kse  ];
      f_SE   = (D.f[DIR_MP0])[knw  ];
      f_BW   = (D.f[DIR_P0P])[kte  ];
      f_TE   = (D.f[DIR_M0M])[kbw  ];
      f_TW   = (D.f[DIR_P0M])[kbe  ];
      f_BE   = (D.f[DIR_M0P])[ktw  ];
      f_BS   = (D.f[DIR_0PP])[ktn  ];
      f_TN   = (D.f[DIR_0MM])[kbs  ];
      f_TS   = (D.f[DIR_0PM])[kbn  ];
      f_BN   = (D.f[DIR_0MP])[kts  ];
      f_BSW  = (D.f[DIR_PPP])[ktne ];
      f_BNE  = (D.f[DIR_MMP])[ktsw ];
      f_BNW  = (D.f[DIR_PMP])[ktse ];
      f_BSE  = (D.f[DIR_MPP])[ktnw ];
      f_TSW  = (D.f[DIR_PPM])[kbne ];
      f_TNE  = (D.f[DIR_MMM])[kbsw ];
      f_TNW  = (D.f[DIR_PMM])[kbse ];
      f_TSE  = (D.f[DIR_MPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW +
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[DIR_000])[kzero]);

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho);


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho);

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho);

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      real om_turb = om1 / (c1o1 + c3o1*om1*max(c0o1, turbViscosity[k_Q[k]]));
      //////////////////////////////////////////////////////////////////////////

      D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, !isEvenTimestep);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Compute incoming f's with zero wall velocity
      ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

      // incoming f's from bounce back
      real f_E_in = 0.0,  f_W_in = 0.0,  f_N_in = 0.0,  f_S_in = 0.0,  f_T_in = 0.0,  f_B_in = 0.0,   f_NE_in = 0.0,  f_SW_in = 0.0,  f_SE_in = 0.0,  f_NW_in = 0.0,  f_TE_in = 0.0,  f_BW_in = 0.0,  f_BE_in = 0.0, f_TW_in = 0.0, f_TN_in = 0.0, f_BS_in = 0.0, f_BN_in = 0.0, f_TS_in = 0.0, f_TNE_in = 0.0, f_TSW_in = 0.0, f_TSE_in = 0.0, f_TNW_in = 0.0, f_BNE_in = 0.0, f_BSW_in = 0.0, f_BSE_in = 0.0, f_BNW_in = 0.0;
      // momentum exchanged with wall at rest
      real wallMomentumX = 0.0, wallMomentumY = 0.0, wallMomentumZ = 0.0;
      real velocityLB = 0.0;
      
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         f_W_in = getInterpolatedDistributionForNoSlipBC(q, f_E, f_W, feq, om_turb);
         wallMomentumX += f_E+f_W_in;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         f_E_in = getInterpolatedDistributionForNoSlipBC(q, f_W, f_E, feq, om_turb);
         wallMomentumX -= f_W+f_E_in;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         f_S_in = getInterpolatedDistributionForNoSlipBC(q, f_N, f_S, feq, om_turb);
         wallMomentumY += f_N+f_S_in;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         f_N_in = getInterpolatedDistributionForNoSlipBC(q, f_S, f_N, feq, om_turb);
         wallMomentumY -= f_S+f_N_in;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         f_B_in = getInterpolatedDistributionForNoSlipBC(q, f_T, f_B, feq, om_turb);
         wallMomentumZ += f_T+f_B_in;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         f_T_in = getInterpolatedDistributionForNoSlipBC(q, f_B, f_T, feq, om_turb);
         wallMomentumZ -= f_B+f_T_in;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_SW_in = getInterpolatedDistributionForNoSlipBC(q, f_NE, f_SW, feq, om_turb);
         wallMomentumX += f_NE+f_SW_in;
         wallMomentumY += f_NE+f_SW_in;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_NE_in = getInterpolatedDistributionForNoSlipBC(q, f_SW, f_NE, feq, om_turb);
         wallMomentumX -= f_SW+f_NE_in;
         wallMomentumY -= f_SW+f_NE_in;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_NW_in = getInterpolatedDistributionForNoSlipBC(q, f_SE, f_NW, feq, om_turb);
         wallMomentumX += f_SE+f_NW_in;
         wallMomentumY -= f_SE+f_NW_in;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_SE_in = getInterpolatedDistributionForNoSlipBC(q, f_NW, f_SE, feq, om_turb);
         wallMomentumX -= f_NW+f_SE_in;
         wallMomentumY += f_NW+f_SE_in;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_BW_in = getInterpolatedDistributionForNoSlipBC(q, f_TE, f_BW, feq, om_turb);
         wallMomentumX += f_TE+f_BW_in;
         wallMomentumZ += f_TE+f_BW_in;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_TE_in = getInterpolatedDistributionForNoSlipBC(q, f_BW, f_TE, feq, om_turb);
         wallMomentumX -= f_BW+f_TE_in;
         wallMomentumZ -= f_BW+f_TE_in;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_TW_in = getInterpolatedDistributionForNoSlipBC(q, f_BE, f_TW, feq, om_turb);
         wallMomentumX += f_BE+f_TW_in;
         wallMomentumZ -= f_BE+f_TW_in;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_BE_in = getInterpolatedDistributionForNoSlipBC(q, f_TW, f_BE, feq, om_turb);
         wallMomentumX -= f_TW+f_BE_in;
         wallMomentumZ += f_TW+f_BE_in;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_BS_in = getInterpolatedDistributionForNoSlipBC(q, f_TN, f_BS, feq, om_turb);
         wallMomentumY += f_TN+f_BS_in;
         wallMomentumZ += f_TN+f_BS_in;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_TN_in = getInterpolatedDistributionForNoSlipBC(q, f_BS, f_TN, feq, om_turb);
         wallMomentumY -= f_BS+f_TN_in;
         wallMomentumZ -= f_BS+f_TN_in;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_TS_in = getInterpolatedDistributionForNoSlipBC(q, f_BN, f_TS, feq, om_turb);
         wallMomentumY += f_BN+f_TS_in;
         wallMomentumZ -= f_BN+f_TS_in;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         f_BN_in = getInterpolatedDistributionForNoSlipBC(q, f_TS, f_BN, feq, om_turb);
         wallMomentumY -= f_TS+f_BN_in;
         wallMomentumZ += f_TS+f_BN_in;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         f_BSW_in = getInterpolatedDistributionForNoSlipBC(q, f_TNE, f_BSW, feq, om_turb);
         wallMomentumX += f_TNE+f_BSW_in;
         wallMomentumY += f_TNE+f_BSW_in;
         wallMomentumZ += f_TNE+f_BSW_in;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         f_TNE_in = getInterpolatedDistributionForNoSlipBC(q, f_BSW, f_TNE, feq, om_turb);
         wallMomentumX -= f_BSW+f_TNE_in;
         wallMomentumY -= f_BSW+f_TNE_in;
         wallMomentumZ -= f_BSW+f_TNE_in;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         f_TSW_in = getInterpolatedDistributionForNoSlipBC(q, f_BNE, f_TSW, feq, om_turb);
         wallMomentumX += f_BNE+f_TSW_in;
         wallMomentumY += f_BNE+f_TSW_in;
         wallMomentumZ -= f_BNE+f_TSW_in;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         f_BNE_in = getInterpolatedDistributionForNoSlipBC(q, f_TSW, f_BNE, feq, om_turb);
         wallMomentumX -= f_TSW+f_BNE_in;
         wallMomentumY -= f_TSW+f_BNE_in;
         wallMomentumZ += f_TSW+f_BNE_in;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         f_BNW_in = getInterpolatedDistributionForNoSlipBC(q, f_TSE, f_BNW, feq, om_turb);
         wallMomentumX += f_TSE+f_BNW_in;
         wallMomentumY -= f_TSE+f_BNW_in;
         wallMomentumZ += f_TSE+f_BNW_in;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         f_TSE_in = getInterpolatedDistributionForNoSlipBC(q, f_BNW, f_TSE, feq, om_turb);
         wallMomentumX -= f_BNW+f_TSE_in;
         wallMomentumY += f_BNW+f_TSE_in;
         wallMomentumZ -= f_BNW+f_TSE_in;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         f_TNW_in = getInterpolatedDistributionForNoSlipBC(q, f_BSE, f_TNW, feq, om_turb);
         wallMomentumX += f_BSE+f_TNW_in;
         wallMomentumY -= f_BSE+f_TNW_in;
         wallMomentumZ -= f_BSE+f_TNW_in;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         f_BSE_in = getInterpolatedDistributionForNoSlipBC(q, f_TNW, f_BSE, feq, om_turb);
         wallMomentumX -= f_TNW+f_BSE_in;
         wallMomentumY += f_TNW+f_BSE_in;
         wallMomentumZ += f_TNW+f_BSE_in;
      }

      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // //Compute wall velocity
      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX=0.0, VeloY=0.0, VeloZ=0.0;

      q = 0.5f;
      real eps = 0.001f;

      iMEM( k, k_N[k],
            normalX, normalY, normalZ,
            vx, vy, vz,
            vx_el,      vy_el,      vz_el,
            vx_w_mean,  vy_w_mean,  vz_w_mean,
            vx1,        vx2,        vx3,
            c1o1+drho,
            samplingOffset,
            q,
            1.0+q,
            eps,
            z0,
            hasWallModelMonitor,
            u_star_monitor,
            wallMomentumX, wallMomentumY, wallMomentumZ,
            VeloX, VeloY, VeloZ);

      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // //Add wall velocity and write f's
      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M00])[kw] = f_W_in - (c6o1*c2o27*( VeloX     ))/(c1o1+q);
         wallMomentumX += -(c6o1*c2o27*( VeloX     ))/(c1o1+q);
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P00])[ke] = f_E_in - (c6o1*c2o27*(-VeloX     ))/(c1o1+q);
         wallMomentumX -= - (c6o1*c2o27*(-VeloX     ))/(c1o1+q);
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0M0])[ks] = f_S_in - (c6o1*c2o27*( VeloY     ))/(c1o1+q);
         wallMomentumY += - (c6o1*c2o27*( VeloY     ))/(c1o1+q);
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0P0])[kn] = f_N_in - (c6o1*c2o27*(-VeloY     ))/(c1o1+q);
         wallMomentumY -=  -(c6o1*c2o27*(-VeloY     ))/(c1o1+q);
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_00M])[kb] = f_B_in - (c6o1*c2o27*( VeloZ     ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c2o27*( VeloZ     ))/(c1o1+q);
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_00P])[kt] = f_T_in - (c6o1*c2o27*(-VeloZ     ))/(c1o1+q);
         wallMomentumZ -= -(c6o1*c2o27*(-VeloZ     ))/(c1o1+q);
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MM0])[ksw] = f_SW_in - (c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
         wallMomentumX +=  -(c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
         wallMomentumY +=  -(c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PP0])[kne] = f_NE_in - (c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
         wallMomentumX -= - (c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
         wallMomentumY -= - (c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MP0])[knw] = f_NW_in - (c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
         wallMomentumX += -(c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
         wallMomentumY -= -(c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PM0])[kse] = f_SE_in - (c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
         wallMomentumX -= - (c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
         wallMomentumY += - (c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M0M])[kbw] = f_BW_in - (c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
         wallMomentumX += - (c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P0P])[kte] = f_TE_in - (c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
         wallMomentumX -= - (c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
         wallMomentumZ -= - (c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M0P])[ktw] = f_TW_in - (c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
         wallMomentumX += - (c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
         wallMomentumZ -= - (c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P0M])[kbe] = f_BE_in - (c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
         wallMomentumX -= - (c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0MM])[kbs] = f_BS_in - (c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
         wallMomentumY += - (c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0PP])[ktn] = f_TN_in - (c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
         wallMomentumY -= - (c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
         wallMomentumZ -= - (c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0MP])[kts] = f_TS_in - (c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
         wallMomentumY += - (c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
         wallMomentumZ -= - (c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0PM])[kbn] = f_BN_in - (c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
         wallMomentumY -= - (c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MMM])[kbsw] = f_BSW_in - (c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
         wallMomentumX += - (c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
         wallMomentumY += - (c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PPP])[ktne] = f_TNE_in - (c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
         wallMomentumX -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
         wallMomentumY -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
         wallMomentumZ -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MMP])[ktsw] = f_TSW_in - (c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
         wallMomentumX += - (c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
         wallMomentumY += - (c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
         wallMomentumZ -= - (c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PPM])[kbne] = f_BNE_in - (c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
         wallMomentumX -= - (c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
         wallMomentumY -= - (c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MPM])[kbnw] = f_BNW_in - (c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
         wallMomentumX += - (c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
         wallMomentumY -= - (c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PMP])[ktse] = f_TSE_in - (c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
         wallMomentumX -= - (c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
         wallMomentumY += - (c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
         wallMomentumZ -= - (c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MPP])[ktnw] = f_TNW_in - (c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
         wallMomentumX += - (c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
         wallMomentumY -= - (c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
         wallMomentumZ -= - (c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PMM])[kbse] = f_BSE_in - (c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
         wallMomentumX -= - (c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
         wallMomentumY += - (c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
         wallMomentumZ += - (c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
      }

      if(hasWallModelMonitor)
      {
         Fx_monitor[k] = wallMomentumX;
         Fy_monitor[k] = wallMomentumY;
         Fz_monitor[k] = wallMomentumZ;
      }

   }
}

//////////////////////////////////////////////////////////////////////////////
__global__ void BBStressDevice27( real* DD,
											            int* k_Q,
                                             int* k_N,
                                             real* QQ,
                                             unsigned int  numberOfBCnodes,
                                             real* vx,
                                             real* vy,
                                             real* vz,
                                             real* normalX,
                                             real* normalY,
                                             real* normalZ,
                                             real* vx_el,
                                             real* vy_el,
                                             real* vz_el,
                                             real* vx_w_mean,
                                             real* vy_w_mean,
                                             real* vz_w_mean,
                                             int* samplingOffset,
                                             real* z0,
                                             bool  hasWallModelMonitor,
                                             real* u_star_monitor,
                                             real* Fx_monitor,
                                             real* Fy_monitor,
                                             real* Fz_monitor,
                                             unsigned int* neighborX,
                                             unsigned int* neighborY,
                                             unsigned int* neighborZ,
                                             unsigned long long numberOfLBnodes,
                                             bool isEvenTimestep)
{

   Distributions27 D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, isEvenTimestep);

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index
   const unsigned  y = blockIdx.x;   // Globaler y-Index
   const unsigned  z = blockIdx.y;   // Globaler z-Index

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB,
         *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
         *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
         *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
         *q_dirBSE, *q_dirBNW;
      q_dirE   = &QQ[DIR_P00 * numberOfBCnodes];
      q_dirW   = &QQ[DIR_M00 * numberOfBCnodes];
      q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];

      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[DIR_P00])[ke   ];
      f_E    = (D.f[DIR_M00])[kw   ];
      f_S    = (D.f[DIR_0P0])[kn   ];
      f_N    = (D.f[DIR_0M0])[ks   ];
      f_B    = (D.f[DIR_00P])[kt   ];
      f_T    = (D.f[DIR_00M])[kb   ];
      f_SW   = (D.f[DIR_PP0])[kne  ];
      f_NE   = (D.f[DIR_MM0])[ksw  ];
      f_NW   = (D.f[DIR_PM0])[kse  ];
      f_SE   = (D.f[DIR_MP0])[knw  ];
      f_BW   = (D.f[DIR_P0P])[kte  ];
      f_TE   = (D.f[DIR_M0M])[kbw  ];
      f_TW   = (D.f[DIR_P0M])[kbe  ];
      f_BE   = (D.f[DIR_M0P])[ktw  ];
      f_BS   = (D.f[DIR_0PP])[ktn  ];
      f_TN   = (D.f[DIR_0MM])[kbs  ];
      f_TS   = (D.f[DIR_0PM])[kbn  ];
      f_BN   = (D.f[DIR_0MP])[kts  ];
      f_BSW  = (D.f[DIR_PPP])[ktne ];
      f_BNE  = (D.f[DIR_MMP])[ktsw ];
      f_BNW  = (D.f[DIR_PMP])[ktse ];
      f_BSE  = (D.f[DIR_MPP])[ktnw ];
      f_TSW  = (D.f[DIR_PPM])[kbne ];
      f_TNE  = (D.f[DIR_MMM])[kbsw ];
      f_TNW  = (D.f[DIR_PMM])[kbse ];
      f_TSE  = (D.f[DIR_MPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW +
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[DIR_000])[kzero]);

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho);


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho);

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////

      D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, !isEvenTimestep);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real f_E_in,  f_W_in,  f_N_in,  f_S_in,  f_T_in,  f_B_in,   f_NE_in,  f_SW_in,  f_SE_in,  f_NW_in,  f_TE_in,  f_BW_in,  f_BE_in,
         f_TW_in, f_TN_in, f_BS_in, f_BN_in, f_TS_in, f_TNE_in, f_TSW_in, f_TSE_in, f_TNW_in, f_BNE_in, f_BSW_in, f_BSE_in, f_BNW_in;

      // momentum exchanged with wall at rest
      real wallMomentumX = 0.0, wallMomentumY = 0.0, wallMomentumZ = 0.0;

      real q;
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_W_in=f_E;
         wallMomentumX += f_E+f_W_in;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_E_in=f_W;
          wallMomentumX -= f_W+f_E_in;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_S_in=f_N;
         wallMomentumY += f_N+f_S_in;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_N_in=f_S;
         wallMomentumY -= f_S+f_N_in;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_B_in=f_T;
         wallMomentumZ += f_T+f_B_in;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_T_in=f_B;
         wallMomentumZ -= f_B+f_T_in;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_SW_in=f_NE;
         wallMomentumX += f_NE+f_SW_in;
         wallMomentumY += f_NE+f_SW_in;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_NE_in=f_SW;
         wallMomentumX -= f_SW+f_NE_in;
         wallMomentumY -= f_SW+f_NE_in;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_NW_in=f_SE;
         wallMomentumX += f_SE+f_NW_in;
         wallMomentumY -= f_SE+f_NW_in;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_SE_in=f_NW;
         wallMomentumX -= f_NW+f_SE_in;
         wallMomentumY += f_NW+f_SE_in;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BW_in=f_TE;
         wallMomentumX += f_TE+f_BW_in;
         wallMomentumZ += f_TE+f_BW_in;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TE_in=f_BW;
         wallMomentumX -= f_BW+f_TE_in;
         wallMomentumZ -= f_BW+f_TE_in;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TW_in=f_BE;
         wallMomentumX += f_BE+f_TW_in;
         wallMomentumZ -= f_BE+f_TW_in;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BE_in=f_TW;
         wallMomentumX -= f_TW+f_BE_in;
         wallMomentumZ += f_TW+f_BE_in;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BS_in=f_TN;
         wallMomentumY += f_TN+f_BS_in;
         wallMomentumZ += f_TN+f_BS_in;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TN_in=f_BS;
         wallMomentumY -= f_BS+f_TN_in;
         wallMomentumZ -= f_BS+f_TN_in;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TS_in=f_BN;
         wallMomentumY += f_BN+f_TS_in;
         wallMomentumZ -= f_BN+f_TS_in;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BN_in=f_TS;
         wallMomentumY -= f_TS+f_BN_in;
         wallMomentumZ += f_TS+f_BN_in;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BSW_in=f_TNE;
         wallMomentumX += f_TNE+f_BSW_in;
         wallMomentumY += f_TNE+f_BSW_in;
         wallMomentumZ += f_TNE+f_BSW_in;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TNE_in=f_BSW;
         wallMomentumX -= f_BSW+f_TNE_in;
         wallMomentumY -= f_BSW+f_TNE_in;
         wallMomentumZ -= f_BSW+f_TNE_in;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TSW_in=f_BNE;
         wallMomentumX += f_BNE+f_TSW_in;
         wallMomentumY += f_BNE+f_TSW_in;
         wallMomentumZ -= f_BNE+f_TSW_in;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BNE_in=f_TSW;
         wallMomentumX -= f_TSW+f_BNE_in;
         wallMomentumY -= f_TSW+f_BNE_in;
         wallMomentumZ += f_TSW+f_BNE_in;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BNW_in=f_TSE;
         wallMomentumX += f_TSE+f_BNW_in;
         wallMomentumY -= f_TSE+f_BNW_in;
         wallMomentumZ += f_TSE+f_BNW_in;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TSE_in=f_BNW;
         wallMomentumX -= f_BNW+f_TSE_in;
         wallMomentumY += f_BNW+f_TSE_in;
         wallMomentumZ -= f_BNW+f_TSE_in;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TNW_in=f_BSE;
         wallMomentumX += f_BSE+f_TNW_in;
         wallMomentumY -= f_BSE+f_TNW_in;
         wallMomentumZ -= f_BSE+f_TNW_in;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BSE_in=f_TNW;
         wallMomentumX -= f_TNW+f_BSE_in;
         wallMomentumY += f_TNW+f_BSE_in;
         wallMomentumZ += f_TNW+f_BSE_in;
      }

      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // //Compute wall velocity
      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX=0.0, VeloY=0.0, VeloZ=0.0;

      q = q_dirB[k];
      real eps = 0.001f;

      iMEM( k, k_N[k],
         normalX, normalY, normalZ,
         vx, vy, vz,
         vx_el,      vy_el,      vz_el,
         vx_w_mean,  vy_w_mean,  vz_w_mean,
         vx1,        vx2,        vx3,
         c1o1+drho,
         samplingOffset,
         q,
         1.0,
         eps,
         z0,
         hasWallModelMonitor,
         u_star_monitor,
         wallMomentumX, wallMomentumY, wallMomentumZ,
         VeloX, VeloY, VeloZ);

      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // //Add wall velocity and write f's
      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M00])[kw] = f_W_in - (c6o1*c2o27*( VeloX     ));
         wallMomentumX += -(c6o1*c2o27*( VeloX     ));
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P00])[ke] = f_E_in - (c6o1*c2o27*(-VeloX     ));
         wallMomentumX -= - (c6o1*c2o27*(-VeloX     ));
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0M0])[ks] = f_S_in - (c6o1*c2o27*( VeloY     ));
         wallMomentumY += - (c6o1*c2o27*( VeloY     ));
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0P0])[kn] = f_N_in - (c6o1*c2o27*(-VeloY     ));
         wallMomentumY -=  -(c6o1*c2o27*(-VeloY     ));
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_00M])[kb] = f_B_in - (c6o1*c2o27*( VeloZ     ));
         wallMomentumZ += - (c6o1*c2o27*( VeloZ     ));
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_00P])[kt] = f_T_in - (c6o1*c2o27*(-VeloZ     ));
         wallMomentumZ -= -(c6o1*c2o27*(-VeloZ     ));
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MM0])[ksw] = f_SW_in - (c6o1*c1o54*(VeloX+VeloY));
         wallMomentumX +=  -(c6o1*c1o54*(VeloX+VeloY));
         wallMomentumY +=  -(c6o1*c1o54*(VeloX+VeloY));
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PP0])[kne] = f_NE_in - (c6o1*c1o54*(-VeloX-VeloY));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX-VeloY));
         wallMomentumY -= - (c6o1*c1o54*(-VeloX-VeloY));
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MP0])[knw] = f_NW_in - (c6o1*c1o54*( VeloX-VeloY));
         wallMomentumX += -(c6o1*c1o54*( VeloX-VeloY));
         wallMomentumY -= -(c6o1*c1o54*( VeloX-VeloY));
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PM0])[kse] = f_SE_in - (c6o1*c1o54*(-VeloX+VeloY));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX+VeloY));
         wallMomentumY += - (c6o1*c1o54*(-VeloX+VeloY));
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M0M])[kbw] = f_BW_in - (c6o1*c1o54*( VeloX+VeloZ));
         wallMomentumX += - (c6o1*c1o54*( VeloX+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( VeloX+VeloZ));
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P0P])[kte] = f_TE_in - (c6o1*c1o54*(-VeloX-VeloZ));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*(-VeloX-VeloZ));
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M0P])[ktw] = f_TW_in - (c6o1*c1o54*( VeloX-VeloZ));
         wallMomentumX += - (c6o1*c1o54*( VeloX-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( VeloX-VeloZ));
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P0M])[kbe] = f_BE_in - (c6o1*c1o54*(-VeloX+VeloZ));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*(-VeloX+VeloZ));
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0MM])[kbs] = f_BS_in - (c6o1*c1o54*( VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o54*( VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( VeloY+VeloZ));
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0PP])[ktn] = f_TN_in - (c6o1*c1o54*( -VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o54*( -VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( -VeloY-VeloZ));
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0MP])[kts] = f_TS_in - (c6o1*c1o54*( VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o54*( VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( VeloY-VeloZ));
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0PM])[kbn] = f_BN_in - (c6o1*c1o54*( -VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o54*( -VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( -VeloY+VeloZ));
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MMM])[kbsw] = f_BSW_in - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PPP])[ktne] = f_TNE_in - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MMP])[ktsw] = f_TSW_in - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PPM])[kbne] = f_BNE_in - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MPM])[kbnw] = f_BNW_in - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PMP])[ktse] = f_TSE_in - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MPP])[ktnw] = f_TNW_in - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PMM])[kbse] = f_BSE_in - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
      }

      if(hasWallModelMonitor)
      {
         Fx_monitor[k] = wallMomentumX;
         Fy_monitor[k] = wallMomentumY;
         Fz_monitor[k] = wallMomentumZ;
      }

   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////
__global__ void BBStressPressureDevice27( real* DD,
											            int* k_Q,
                                             int* k_N,
                                             real* QQ,
                                             unsigned int  numberOfBCnodes,
                                             real* vx,
                                             real* vy,
                                             real* vz,
                                             real* normalX,
                                             real* normalY,
                                             real* normalZ,
                                             real* vx_el,
                                             real* vy_el,
                                             real* vz_el,
                                             real* vx_w_mean,
                                             real* vy_w_mean,
                                             real* vz_w_mean,
                                             int* samplingOffset,
                                             real* z0,
                                             bool  hasWallModelMonitor,
                                             real* u_star_monitor,
                                             real* Fx_monitor,
                                             real* Fy_monitor,
                                             real* Fz_monitor,
                                             unsigned int* neighborX,
                                             unsigned int* neighborY,
                                             unsigned int* neighborZ,
                                             unsigned long long numberOfLBnodes,
                                             bool isEvenTimestep)
{
   Distributions27 D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, isEvenTimestep);

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index
   const unsigned  y = blockIdx.x;   // Globaler y-Index
   const unsigned  z = blockIdx.y;   // Globaler z-Index

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB,
         *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
         *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
         *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
         *q_dirBSE, *q_dirBNW;
      q_dirE   = &QQ[DIR_P00 * numberOfBCnodes];
      q_dirW   = &QQ[DIR_M00 * numberOfBCnodes];
      q_dirN   = &QQ[DIR_0P0 * numberOfBCnodes];
      q_dirS   = &QQ[DIR_0M0 * numberOfBCnodes];
      q_dirT   = &QQ[DIR_00P * numberOfBCnodes];
      q_dirB   = &QQ[DIR_00M * numberOfBCnodes];
      q_dirNE  = &QQ[DIR_PP0 * numberOfBCnodes];
      q_dirSW  = &QQ[DIR_MM0 * numberOfBCnodes];
      q_dirSE  = &QQ[DIR_PM0 * numberOfBCnodes];
      q_dirNW  = &QQ[DIR_MP0 * numberOfBCnodes];
      q_dirTE  = &QQ[DIR_P0P * numberOfBCnodes];
      q_dirBW  = &QQ[DIR_M0M * numberOfBCnodes];
      q_dirBE  = &QQ[DIR_P0M * numberOfBCnodes];
      q_dirTW  = &QQ[DIR_M0P * numberOfBCnodes];
      q_dirTN  = &QQ[DIR_0PP * numberOfBCnodes];
      q_dirBS  = &QQ[DIR_0MM * numberOfBCnodes];
      q_dirBN  = &QQ[DIR_0PM * numberOfBCnodes];
      q_dirTS  = &QQ[DIR_0MP * numberOfBCnodes];
      q_dirTNE = &QQ[DIR_PPP * numberOfBCnodes];
      q_dirTSW = &QQ[DIR_MMP * numberOfBCnodes];
      q_dirTSE = &QQ[DIR_PMP * numberOfBCnodes];
      q_dirTNW = &QQ[DIR_MPP * numberOfBCnodes];
      q_dirBNE = &QQ[DIR_PPM * numberOfBCnodes];
      q_dirBSW = &QQ[DIR_MMM * numberOfBCnodes];
      q_dirBSE = &QQ[DIR_PMM * numberOfBCnodes];
      q_dirBNW = &QQ[DIR_MPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];

      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[DIR_P00])[ke   ];
      f_E    = (D.f[DIR_M00])[kw   ];
      f_S    = (D.f[DIR_0P0])[kn   ];
      f_N    = (D.f[DIR_0M0])[ks   ];
      f_B    = (D.f[DIR_00P])[kt   ];
      f_T    = (D.f[DIR_00M])[kb   ];
      f_SW   = (D.f[DIR_PP0])[kne  ];
      f_NE   = (D.f[DIR_MM0])[ksw  ];
      f_NW   = (D.f[DIR_PM0])[kse  ];
      f_SE   = (D.f[DIR_MP0])[knw  ];
      f_BW   = (D.f[DIR_P0P])[kte  ];
      f_TE   = (D.f[DIR_M0M])[kbw  ];
      f_TW   = (D.f[DIR_P0M])[kbe  ];
      f_BE   = (D.f[DIR_M0P])[ktw  ];
      f_BS   = (D.f[DIR_0PP])[ktn  ];
      f_TN   = (D.f[DIR_0MM])[kbs  ];
      f_TS   = (D.f[DIR_0PM])[kbn  ];
      f_BN   = (D.f[DIR_0MP])[kts  ];
      f_BSW  = (D.f[DIR_PPP])[ktne ];
      f_BNE  = (D.f[DIR_MMP])[ktsw ];
      f_BNW  = (D.f[DIR_PMP])[ktse ];
      f_BSE  = (D.f[DIR_MPP])[ktnw ];
      f_TSW  = (D.f[DIR_PPM])[kbne ];
      f_TNE  = (D.f[DIR_MMM])[kbsw ];
      f_TNW  = (D.f[DIR_PMM])[kbse ];
      f_TSE  = (D.f[DIR_MPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW +
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[DIR_000])[kzero]);

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho);


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho);

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real f_E_in,  f_W_in,  f_N_in,  f_S_in,  f_T_in,  f_B_in,   f_NE_in,  f_SW_in,  f_SE_in,  f_NW_in,  f_TE_in,  f_BW_in,  f_BE_in,
         f_TW_in, f_TN_in, f_BS_in, f_BN_in, f_TS_in, f_TNE_in, f_TSW_in, f_TSE_in, f_TNW_in, f_BNE_in, f_BSW_in, f_BSE_in, f_BNW_in;

      // momentum exchanged with wall at rest
      real wallMomentumX = 0.0, wallMomentumY = 0.0, wallMomentumZ = 0.0;

      real q;
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_W_in=f_E - c2o27 * drho;
         wallMomentumX += f_E+f_W_in;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_E_in=f_W - c2o27 * drho;
          wallMomentumX -= f_W+f_E_in;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_S_in=f_N - c2o27 * drho;
         wallMomentumY += f_N+f_S_in;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_N_in=f_S - c2o27 * drho;
         wallMomentumY -= f_S+f_N_in;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_B_in=f_T - c2o27 * drho;
         wallMomentumZ += f_T+f_B_in;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_T_in=f_B - c2o27 * drho;
         wallMomentumZ -= f_B+f_T_in;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_SW_in=f_NE - c1o54 * drho;
         wallMomentumX += f_NE+f_SW_in;
         wallMomentumY += f_NE+f_SW_in;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_NE_in=f_SW - c1o54 * drho;
         wallMomentumX -= f_SW+f_NE_in;
         wallMomentumY -= f_SW+f_NE_in;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_NW_in=f_SE - c1o54 * drho;
         wallMomentumX += f_SE+f_NW_in;
         wallMomentumY -= f_SE+f_NW_in;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_SE_in=f_NW - c1o54 * drho;
         wallMomentumX -= f_NW+f_SE_in;
         wallMomentumY += f_NW+f_SE_in;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BW_in=f_TE - c1o54 * drho;
         wallMomentumX += f_TE+f_BW_in;
         wallMomentumZ += f_TE+f_BW_in;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TE_in=f_BW - c1o54 * drho;
         wallMomentumX -= f_BW+f_TE_in;
         wallMomentumZ -= f_BW+f_TE_in;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TW_in=f_BE - c1o54 * drho;
         wallMomentumX += f_BE+f_TW_in;
         wallMomentumZ -= f_BE+f_TW_in;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BE_in=f_TW - c1o54 * drho;
         wallMomentumX -= f_TW+f_BE_in;
         wallMomentumZ += f_TW+f_BE_in;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BS_in=f_TN - c1o54 * drho;
         wallMomentumY += f_TN+f_BS_in;
         wallMomentumZ += f_TN+f_BS_in;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TN_in=f_BS - c1o54 * drho;
         wallMomentumY -= f_BS+f_TN_in;
         wallMomentumZ -= f_BS+f_TN_in;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TS_in=f_BN - c1o54 * drho;
         wallMomentumY += f_BN+f_TS_in;
         wallMomentumZ -= f_BN+f_TS_in;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BN_in=f_TS - c1o54 * drho;
         wallMomentumY -= f_TS+f_BN_in;
         wallMomentumZ += f_TS+f_BN_in;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BSW_in=f_TNE - c1o216 * drho;
         wallMomentumX += f_TNE+f_BSW_in;
         wallMomentumY += f_TNE+f_BSW_in;
         wallMomentumZ += f_TNE+f_BSW_in;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TNE_in=f_BSW - c1o216 * drho;
         wallMomentumX -= f_BSW+f_TNE_in;
         wallMomentumY -= f_BSW+f_TNE_in;
         wallMomentumZ -= f_BSW+f_TNE_in;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TSW_in=f_BNE - c1o216 * drho;
         wallMomentumX += f_BNE+f_TSW_in;
         wallMomentumY += f_BNE+f_TSW_in;
         wallMomentumZ -= f_BNE+f_TSW_in;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BNE_in=f_TSW - c1o216 * drho;
         wallMomentumX -= f_TSW+f_BNE_in;
         wallMomentumY -= f_TSW+f_BNE_in;
         wallMomentumZ += f_TSW+f_BNE_in;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BNW_in=f_TSE - c1o216 * drho;
         wallMomentumX += f_TSE+f_BNW_in;
         wallMomentumY -= f_TSE+f_BNW_in;
         wallMomentumZ += f_TSE+f_BNW_in;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TSE_in=f_BNW - c1o216 * drho;
         wallMomentumX -= f_BNW+f_TSE_in;
         wallMomentumY += f_BNW+f_TSE_in;
         wallMomentumZ -= f_BNW+f_TSE_in;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TNW_in=f_BSE - c1o216 * drho;
         wallMomentumX += f_BSE+f_TNW_in;
         wallMomentumY -= f_BSE+f_TNW_in;
         wallMomentumZ -= f_BSE+f_TNW_in;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BSE_in=f_TNW - c1o216 * drho;
         wallMomentumX -= f_TNW+f_BSE_in;
         wallMomentumY += f_TNW+f_BSE_in;
         wallMomentumZ += f_TNW+f_BSE_in;
      }

      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // //Compute wall velocity
      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX=0.0, VeloY=0.0, VeloZ=0.0;

      q = q_dirB[k];
      real eps = 0.001f;

      iMEM( k, k_N[k],
         normalX, normalY, normalZ,
         vx, vy, vz,
         vx_el,      vy_el,      vz_el,
         vx_w_mean,  vy_w_mean,  vz_w_mean,
         vx1,        vx2,        vx3,
         c1o1+drho,
         samplingOffset,
         q,
         1.0,
         eps,
         z0,
         hasWallModelMonitor,
         u_star_monitor,
         wallMomentumX, wallMomentumY, wallMomentumZ,
         VeloX, VeloY, VeloZ);

      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // //Add wall velocity and write f's
      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M00])[kw] = f_W_in - (c6o1*c2o27*( VeloX     ));
         wallMomentumX += -(c6o1*c2o27*( VeloX     ));
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P00])[ke] = f_E_in - (c6o1*c2o27*(-VeloX     ));
         wallMomentumX -= - (c6o1*c2o27*(-VeloX     ));
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0M0])[ks] = f_S_in - (c6o1*c2o27*( VeloY     ));
         wallMomentumY += - (c6o1*c2o27*( VeloY     ));
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0P0])[kn] = f_N_in - (c6o1*c2o27*(-VeloY     ));
         wallMomentumY -=  -(c6o1*c2o27*(-VeloY     ));
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_00M])[kb] = f_B_in - (c6o1*c2o27*( VeloZ     ));
         wallMomentumZ += - (c6o1*c2o27*( VeloZ     ));
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_00P])[kt] = f_T_in - (c6o1*c2o27*(-VeloZ     ));
         wallMomentumZ -= -(c6o1*c2o27*(-VeloZ     ));
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MM0])[ksw] = f_SW_in - (c6o1*c1o54*(VeloX+VeloY));
         wallMomentumX +=  -(c6o1*c1o54*(VeloX+VeloY));
         wallMomentumY +=  -(c6o1*c1o54*(VeloX+VeloY));
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PP0])[kne] = f_NE_in - (c6o1*c1o54*(-VeloX-VeloY));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX-VeloY));
         wallMomentumY -= - (c6o1*c1o54*(-VeloX-VeloY));
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MP0])[knw] = f_NW_in - (c6o1*c1o54*( VeloX-VeloY));
         wallMomentumX += -(c6o1*c1o54*( VeloX-VeloY));
         wallMomentumY -= -(c6o1*c1o54*( VeloX-VeloY));
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PM0])[kse] = f_SE_in - (c6o1*c1o54*(-VeloX+VeloY));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX+VeloY));
         wallMomentumY += - (c6o1*c1o54*(-VeloX+VeloY));
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M0M])[kbw] = f_BW_in - (c6o1*c1o54*( VeloX+VeloZ));
         wallMomentumX += - (c6o1*c1o54*( VeloX+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( VeloX+VeloZ));
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P0P])[kte] = f_TE_in - (c6o1*c1o54*(-VeloX-VeloZ));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*(-VeloX-VeloZ));
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_M0P])[ktw] = f_TW_in - (c6o1*c1o54*( VeloX-VeloZ));
         wallMomentumX += - (c6o1*c1o54*( VeloX-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( VeloX-VeloZ));
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_P0M])[kbe] = f_BE_in - (c6o1*c1o54*(-VeloX+VeloZ));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*(-VeloX+VeloZ));
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0MM])[kbs] = f_BS_in - (c6o1*c1o54*( VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o54*( VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( VeloY+VeloZ));
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0PP])[ktn] = f_TN_in - (c6o1*c1o54*( -VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o54*( -VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( -VeloY-VeloZ));
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0MP])[kts] = f_TS_in - (c6o1*c1o54*( VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o54*( VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( VeloY-VeloZ));
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_0PM])[kbn] = f_BN_in - (c6o1*c1o54*( -VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o54*( -VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( -VeloY+VeloZ));
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MMM])[kbsw] = f_BSW_in - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PPP])[ktne] = f_TNE_in - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MMP])[ktsw] = f_TSW_in - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PPM])[kbne] = f_BNE_in - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MPM])[kbnw] = f_BNW_in - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PMP])[ktse] = f_TSE_in - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_MPP])[ktnw] = f_TNW_in - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[DIR_PMM])[kbse] = f_BSE_in - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
      }

      if(hasWallModelMonitor)
      {
         Fx_monitor[k] = wallMomentumX;
         Fy_monitor[k] = wallMomentumY;
         Fz_monitor[k] = wallMomentumZ;
      }

   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////