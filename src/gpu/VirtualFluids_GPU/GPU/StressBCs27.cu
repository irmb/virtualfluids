#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;

//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QStressDeviceComp27(real* DD, 
											   int* k_Q, 
                                    int* k_N, 
											   real* QQ,
                                    unsigned int sizeQ,
                                    real om1, 
                                    real* vx,
                                    real* vy,
                                    real* vz,
                                    real* normalX,
                                    real* normalY,
                                    real* normalZ,
											   unsigned int* neighborX,
                                    unsigned int* neighborY,
                                    unsigned int* neighborZ,
                                    unsigned int size_Mat, 
                                    bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)//get right array of post coll f's
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ/*kQ*/)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;      //get right adress of post-coll f's
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dirE   ])[ke   ];     //post-coll f's
      f_E    = (D.f[dirW   ])[kw   ];
      f_S    = (D.f[dirN   ])[kn   ];
      f_N    = (D.f[dirS   ])[ks   ];
      f_B    = (D.f[dirT   ])[kt   ];
      f_T    = (D.f[dirB   ])[kb   ];
      f_SW   = (D.f[dirNE  ])[kne  ];
      f_NE   = (D.f[dirSW  ])[ksw  ];
      f_NW   = (D.f[dirSE  ])[kse  ];
      f_SE   = (D.f[dirNW  ])[knw  ];
      f_BW   = (D.f[dirTE  ])[kte  ];
      f_TE   = (D.f[dirBW  ])[kbw  ];
      f_TW   = (D.f[dirBE  ])[kbe  ];
      f_BE   = (D.f[dirTW  ])[ktw  ];
      f_BS   = (D.f[dirTN  ])[ktn  ];
      f_TN   = (D.f[dirBS  ])[kbs  ];
      f_TS   = (D.f[dirBN  ])[kbn  ];
      f_BN   = (D.f[dirTS  ])[kts  ];
      f_BSW  = (D.f[dirTNE ])[ktne ];
      f_BNE  = (D.f[dirTSW ])[ktsw ];
      f_BNW  = (D.f[dirTSE ])[ktse ];
      f_BSE  = (D.f[dirTNW ])[ktnw ];
      f_TSW  = (D.f[dirBNE ])[kbne ];
      f_TNE  = (D.f[dirBSW ])[kbsw ];
      f_TNW  = (D.f[dirBSE ])[kbse ];
      f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)      //get adress where incoming f's should be written to
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Compute incoming f's with zero wall velocity
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX=0.057735, VeloY=0.0, VeloZ=0.0; 
      
      // if(k==3071){ // 3071 9120 9215
      //    printf("======================================\n");
      //    printf("k \t %u\n", k);
      //    printf("E \t %f \n", q_dirE[k]);
      //    printf("NE \t %f \n", q_dirNE[k]);
      //    printf("W \t %f \n", q_dirW[k]);
      //    printf("NW \t %f \n", q_dirNW[k]);
      //    printf("N \t %f \n", q_dirN[k]);
      //    printf("S \t %f \n", q_dirS[k]);
      //    printf("SE \t %f \n", q_dirSE[k]);
      //    printf("NE \t %f \n", q_dirNE[k]);
      //    printf("SW \t %f \n", q_dirSW[k]);
      //    printf("B \t %f \n", q_dirB[k]);
      //    printf("BS \t %f \n", q_dirBS[k]);
      //    printf("BN \t %f \n", q_dirBN[k]);
      //    printf("BW \t %f \n", q_dirBW[k]);
      //    printf("BE \t %f \n", q_dirBE[k]);
      //    printf("BNE \t %f \n", q_dirBNE[k]);
      //    printf("BNW \t %f \n", q_dirBNW[k]);
      //    printf("BSE \t %f \n", q_dirBSE[k]);
      //    printf("BSW \t %f \n", q_dirBSW[k]);
      //    printf("T \t %f \n", q_dirT[k]);
      //    printf("TS \t %f \n", q_dirTS[k]);
      //    printf("TN \t %f \n", q_dirTN[k]);
      //    printf("TW \t %f \n", q_dirTW[k]);
      //    printf("TE \t %f \n", q_dirTE[k]);
      //    printf("TNE \t %f \n", q_dirTNE[k]);
      //    printf("TNW \t %f \n", q_dirTNW[k]);
      //    printf("TSE \t %f \n", q_dirTSE[k]);
      //    printf("TSW \t %f \n\n", q_dirTSW[k]);
      // }

      //ToDo anders Klammern
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirW])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirE])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirS])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirN])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirB])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirT])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
         //(D.f[dirBSE])[kbse]=zero;
      }

      // incoming f's from bounce back
      // real f_E_in,  f_W_in,  f_N_in,  f_S_in,  f_T_in,  f_B_in,   f_NE_in,  f_SW_in,  f_SE_in,  f_NW_in,  f_TE_in,  f_BW_in,  f_BE_in,
      //    f_TW_in, f_TN_in, f_BS_in, f_BN_in, f_TS_in, f_TNE_in, f_TSW_in, f_TSE_in, f_TNW_in, f_BNE_in, f_BSW_in, f_BSE_in, f_BNW_in;
      // // momentum exchanged with wall at rest
      // real wallMomentumX = 0.0, wallMomentumY = 0.0, wallMomentumZ = 0.0;

      // q = q_dirE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
      //    f_W_in=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W))/(c1o1+q) - c2o27 * drho;
      //    wallMomentumX += f_E+f_W_in;
      // }

      // q = q_dirW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
      //    f_E_in=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E))/(c1o1+q) - c2o27 * drho;
      //    wallMomentumX -= f_W+f_E_in;
      // }

      // q = q_dirN[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
      //    f_S_in=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S))/(c1o1+q) - c2o27 * drho;
      //    wallMomentumY += f_N+f_S_in;
      // }

      // q = q_dirS[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
      //    f_N_in=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N))/(c1o1+q) - c2o27 * drho;
      //    wallMomentumY -= f_S+f_N_in;
      // }

      // q = q_dirT[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
      //    f_B_in=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B))/(c1o1+q) - c2o27 * drho;
      //    wallMomentumZ += f_T+f_B_in;
      // }

      // q = q_dirB[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
      //    f_T_in=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T))/(c1o1+q) - c2o27 * drho;
      //    wallMomentumZ -= f_B+f_T_in;
      // }

      // q = q_dirNE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
      //    f_SW_in=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumX += f_NE+f_SW_in;
      //    wallMomentumY += f_NE+f_SW_in;
      // }

      // q = q_dirSW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
      //    f_NE_in=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumX -= f_SW+f_NE_in;
      //    wallMomentumY -= f_SW+f_NE_in;
      // }

      // q = q_dirSE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
      //    f_NW_in=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumX += f_SE+f_NW_in;
      //    wallMomentumY -= f_SE+f_NW_in;
      // }

      // q = q_dirNW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
      //    f_SE_in=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumX -= f_NW+f_SE_in;
      //    wallMomentumY += f_NW+f_SE_in;
      // }

      // q = q_dirTE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
      //    f_BW_in=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumX += f_TE+f_BW_in;
      //    wallMomentumZ += f_TE+f_BW_in;
      // }

      // q = q_dirBW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
      //    f_TE_in=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumX -= f_BW+f_TE_in;
      //    wallMomentumZ -= f_BW+f_TE_in;
      // }

      // q = q_dirBE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
      //    f_TW_in=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumX += f_BE+f_TW_in;
      //    wallMomentumZ -= f_BE+f_TW_in;
      // }

      // q = q_dirTW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
      //    f_BE_in=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumX -= f_TW+f_BE_in;
      //    wallMomentumZ += f_TW+f_BE_in;
      // }

      // q = q_dirTN[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
      //    f_BS_in=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumY += f_TN+f_BS_in;
      //    wallMomentumZ += f_TN+f_BS_in;
      // }

      // q = q_dirBS[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
      //    f_TN_in=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumY -= f_BS+f_TN_in;
      //    wallMomentumZ -= f_BS+f_TN_in;
      // }

      // q = q_dirBN[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
      //    f_TS_in=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumY += f_BN+f_TS_in;
      //    wallMomentumZ -= f_BN+f_TS_in;
      // }

      // q = q_dirTS[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
      //    f_BN_in=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN))/(c1o1+q) - c1o54 * drho;
      //    wallMomentumY -= f_TS+f_BN_in;
      //    wallMomentumZ += f_TS+f_BN_in;
      // }

      // q = q_dirTNE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
      //    f_BSW_in=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q) - c1o216 * drho;
      //    wallMomentumX += f_TNE+f_BSW_in;
      //    wallMomentumY += f_TNE+f_BSW_in;
      //    wallMomentumZ += f_TNE+f_BSW_in;
      // }

      // q = q_dirBSW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
      //    f_TNE_in=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q) - c1o216 * drho;
      //    wallMomentumX -= f_BSW+f_TNE_in;
      //    wallMomentumY -= f_BSW+f_TNE_in;
      //    wallMomentumZ -= f_BSW+f_TNE_in;
      // }

      // q = q_dirBNE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
      //    f_TSW_in=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q) - c1o216 * drho;
      //    wallMomentumX += f_BNE+f_TSW_in;
      //    wallMomentumY += f_BNE+f_TSW_in;
      //    wallMomentumZ -= f_BNE+f_TSW_in;
      // }

      // q = q_dirTSW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
      //    f_BNE_in=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q) - c1o216 * drho;
      //    wallMomentumX -= f_TSW+f_BNE_in;
      //    wallMomentumY -= f_TSW+f_BNE_in;
      //    wallMomentumZ += f_TSW+f_BNE_in;
      // }

      // q = q_dirTSE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
      //    f_BNW_in=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q) - c1o216 * drho;
      //    wallMomentumX += f_TSE+f_BNW_in;
      //    wallMomentumY -= f_TSE+f_BNW_in;
      //    wallMomentumZ += f_TSE+f_BNW_in;
      // }

      // q = q_dirBNW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
      //    f_TSE_in=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q) - c1o216 * drho;
      //    wallMomentumX -= f_BNW+f_TSE_in;
      //    wallMomentumY += f_BNW+f_TSE_in;
      //    wallMomentumZ -= f_BNW+f_TSE_in;
      // }

      // q = q_dirBSE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
      //    f_TNW_in=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q) - c1o216 * drho;
      //    wallMomentumX += f_BSE+f_TNW_in;
      //    wallMomentumY -= f_BSE+f_TNW_in;
      //    wallMomentumZ -= f_BSE+f_TNW_in;
      // }

      // q = q_dirTNW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
      //    f_BSE_in=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q) - c1o216 * drho;
      //    wallMomentumX -= f_TNW+f_BSE_in;
      //    wallMomentumY += f_TNW+f_BSE_in;
      //    wallMomentumZ += f_TNW+f_BSE_in;
      // }

      // // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // // //Compute wall velocity
      // // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // real VeloX=0.1, VeloY=0.0, VeloZ=0.0; 

      // // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // // //Add wall velocity and write f's
      // // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

      // q = q_dirE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirW])[kw] = f_W_in - (c6o1*c2o27*( VeloX     ))/(c1o1+q);
      // }

      // q = q_dirW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirE])[ke] = f_E_in - (c6o1*c2o27*(-VeloX     ))/(c1o1+q);
      // }

      // q = q_dirN[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirS])[ks] = f_S_in - (c6o1*c2o27*( VeloY     ))/(c1o1+q);
      // }

      // q = q_dirS[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirN])[kn] = f_N_in - (c6o1*c2o27*(-VeloY     ))/(c1o1+q);
      // }

      // q = q_dirT[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirB])[kb] = f_B_in - (c6o1*c2o27*( VeloZ     ))/(c1o1+q);
      // }

      // q = q_dirB[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirT])[kt] = f_T_in - (c6o1*c2o27*(-VeloZ     ))/(c1o1+q);
      // }

      // q = q_dirNE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirSW])[ksw] = f_SW_in - (c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
      // }

      // q = q_dirSW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirNE])[kne] = f_NE_in - (c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
      // }

      // q = q_dirSE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirNW])[knw] = f_NW_in - (c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
      // }

      // q = q_dirNW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirSE])[kse] = f_SE_in - (c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
      // }

      // q = q_dirTE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirBW])[kbw] = f_BW_in - (c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q); 
      // }

      // q = q_dirBW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirTE])[kte] = f_TE_in - (c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
      // }

      // q = q_dirBE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirTW])[ktw] = f_TW_in - (c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
      // }

      // q = q_dirTW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirBE])[kbe] = f_BE_in - (c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
      // }

      // q = q_dirTN[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirBS])[kbs] = f_BS_in - (c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
      // }

      // q = q_dirBS[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirTN])[ktn] = f_TN_in - (c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
      // }

      // q = q_dirBN[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirTS])[kts] = f_TS_in - (c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
      // }

      // q = q_dirTS[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirBN])[kbn] = f_BN_in - (c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
      // }

      // q = q_dirTNE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirBSW])[kbsw] = f_BSW_in - (c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
      // }

      // q = q_dirBSW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirTNE])[ktne] = f_TNE_in - (c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
      // }

      // q = q_dirBNE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirTSW])[ktsw] = f_TSW_in - (c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
      // }

      // q = q_dirTSW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirBNE])[kbne] = f_BNE_in - (c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
      // }

      // q = q_dirTSE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirBNW])[kbnw] = f_BNW_in - (c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
      // }

      // q = q_dirBNW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirTSE])[ktse] = f_TSE_in - (c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
      // }

      // q = q_dirBSE[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirTNW])[ktnw] = f_TNW_in - (c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
      // }

      // q = q_dirTNW[k];
      // if (q>=c0o1 && q<=c1o1)
      // {
      //    (D.f[dirBSE])[kbse] = f_BSE_in - (c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
      // }

   }
}