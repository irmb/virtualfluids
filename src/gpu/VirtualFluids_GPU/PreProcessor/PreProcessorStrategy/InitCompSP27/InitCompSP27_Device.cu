#include "hip/hip_runtime.h"
#include "LBM/LB.h" 
#include "LBM/D3Q27.h"
#include <lbm/constants/NumericConstants.h>

using namespace vf::lbm::constant;
#include "math.h"

#include <stdio.h>

extern "C" __global__ void LB_Init_Comp_SP_27(unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int* geoD,
	real* rho,
	real* ux,
	real* uy,
	real* uz,
	unsigned int size_Mat,
	real* DD,
	bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   geoD[k];

      if( BC != GEO_SOLID &&  BC != GEO_VOID)
      {
         Distributions27 D;
         if (EvenOrOdd==true)
         {
            D.f[E   ] = &DD[E   *size_Mat];
            D.f[W   ] = &DD[W   *size_Mat];
            D.f[N   ] = &DD[N   *size_Mat];
            D.f[S   ] = &DD[S   *size_Mat];
            D.f[T   ] = &DD[T   *size_Mat];
            D.f[B   ] = &DD[B   *size_Mat];
            D.f[NE  ] = &DD[NE  *size_Mat];
            D.f[SW  ] = &DD[SW  *size_Mat];
            D.f[SE  ] = &DD[SE  *size_Mat];
            D.f[NW  ] = &DD[NW  *size_Mat];
            D.f[TE  ] = &DD[TE  *size_Mat];
            D.f[BW  ] = &DD[BW  *size_Mat];
            D.f[BE  ] = &DD[BE  *size_Mat];
            D.f[TW  ] = &DD[TW  *size_Mat];
            D.f[TN  ] = &DD[TN  *size_Mat];
            D.f[BS  ] = &DD[BS  *size_Mat];
            D.f[BN  ] = &DD[BN  *size_Mat];
            D.f[TS  ] = &DD[TS  *size_Mat];
            D.f[dirREST] = &DD[dirREST*size_Mat];
            D.f[TNE ] = &DD[TNE *size_Mat];
            D.f[TSW ] = &DD[TSW *size_Mat];
            D.f[TSE ] = &DD[TSE *size_Mat];
            D.f[TNW ] = &DD[TNW *size_Mat];
            D.f[BNE ] = &DD[BNE *size_Mat];
            D.f[BSW ] = &DD[BSW *size_Mat];
            D.f[BSE ] = &DD[BSE *size_Mat];
            D.f[BNW ] = &DD[BNW *size_Mat];
         }
         else
         {
            D.f[W   ] = &DD[E   *size_Mat];
            D.f[E   ] = &DD[W   *size_Mat];
            D.f[S   ] = &DD[N   *size_Mat];
            D.f[N   ] = &DD[S   *size_Mat];
            D.f[B   ] = &DD[T   *size_Mat];
            D.f[T   ] = &DD[B   *size_Mat];
            D.f[SW  ] = &DD[NE  *size_Mat];
            D.f[NE  ] = &DD[SW  *size_Mat];
            D.f[NW  ] = &DD[SE  *size_Mat];
            D.f[SE  ] = &DD[NW  *size_Mat];
            D.f[BW  ] = &DD[TE  *size_Mat];
            D.f[TE  ] = &DD[BW  *size_Mat];
            D.f[TW  ] = &DD[BE  *size_Mat];
            D.f[BE  ] = &DD[TW  *size_Mat];
            D.f[BS  ] = &DD[TN  *size_Mat];
            D.f[TN  ] = &DD[BS  *size_Mat];
            D.f[TS  ] = &DD[BN  *size_Mat];
            D.f[BN  ] = &DD[TS  *size_Mat];
            D.f[dirREST] = &DD[dirREST*size_Mat];
            D.f[BSW ] = &DD[TNE *size_Mat];
            D.f[BNE ] = &DD[TSW *size_Mat];
            D.f[BNW ] = &DD[TSE *size_Mat];
            D.f[BSE ] = &DD[TNW *size_Mat];
            D.f[TSW ] = &DD[BNE *size_Mat];
            D.f[TNE ] = &DD[BSW *size_Mat];
            D.f[TNW ] = &DD[BSE *size_Mat];
            D.f[TSE ] = &DD[BNW *size_Mat];
         }
         //////////////////////////////////////////////////////////////////////////
         real drho = rho[k];//0.0f;//
         real  vx1 = ux[k]; //0.0f;//
         real  vx2 = uy[k]; //0.0f;//
         real  vx3 = uz[k]; //0.0f;//
         //////////////////////////////////////////////////////////////////////////
         //index
         //////////////////////////////////////////////////////////////////////////
         unsigned int kzero= k;
         unsigned int ke   = k;
         unsigned int kw   = neighborX[k];
         unsigned int kn   = k;
         unsigned int ks   = neighborY[k];
         unsigned int kt   = k;
         unsigned int kb   = neighborZ[k];
         unsigned int ksw  = neighborY[kw];
         unsigned int kne  = k;
         unsigned int kse  = ks;
         unsigned int knw  = kw;
         unsigned int kbw  = neighborZ[kw];
         unsigned int kte  = k;
         unsigned int kbe  = kb;
         unsigned int ktw  = kw;
         unsigned int kbs  = neighborZ[ks];
         unsigned int ktn  = k;
         unsigned int kbn  = kb;
         unsigned int kts  = ks;
         unsigned int ktse = ks;
         unsigned int kbnw = kbw;
         unsigned int ktnw = kw;
         unsigned int kbse = kbs;
         unsigned int ktsw = ksw;
         unsigned int kbne = kb;
         unsigned int ktne = k;
         unsigned int kbsw = neighborZ[ksw];
         //////////////////////////////////////////////////////////////////////////
         real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

         (D.f[dirREST])[kzero] =   c8o27* (drho-cu_sq*(c1o1+drho));
         (D.f[E   ])[ke   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq));
         (D.f[W   ])[kw   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq));
         (D.f[N   ])[kn   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq));
         (D.f[S   ])[ks   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
         (D.f[T   ])[kt   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq));
         (D.f[B   ])[kb   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq));
         (D.f[NE  ])[kne  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
         (D.f[SW  ])[ksw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
         (D.f[SE  ])[kse  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
         (D.f[NW  ])[knw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
         (D.f[TE  ])[kte  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
         (D.f[BW  ])[kbw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
         (D.f[BE  ])[kbe  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
         (D.f[TW  ])[ktw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
         (D.f[TN  ])[ktn  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
         (D.f[BS  ])[kbs  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
         (D.f[BN  ])[kbn  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
         (D.f[TS  ])[kts  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
         (D.f[TNE ])[ktne ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
         (D.f[BSW ])[kbsw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
         (D.f[BNE ])[kbne ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
         (D.f[TSW ])[ktsw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
         (D.f[TSE ])[ktse ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
         (D.f[BNW ])[kbnw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
         (D.f[BSE ])[kbse ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
         (D.f[TNW ])[ktnw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));
      }
   }
}










////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LB_Init_Comp_Neq_SP_27( unsigned int* neighborX,
                                                   unsigned int* neighborY,
                                                   unsigned int* neighborZ,
                                                   unsigned int* neighborWSB,
                                                   unsigned int* geoD,
                                                   real* rho,
                                                   real* ux,
                                                   real* uy,
                                                   real* uz,
                                                   unsigned int size_Mat,
                                                   real* DD,
                                                   real omega,
                                                   bool EvenOrOdd)
{
    ////////////////////////////////////////////////////////////////////////////////
    const unsigned  x = threadIdx.x;  // Globaler x-Index 
    const unsigned  y = blockIdx.x;   // Globaler y-Index 
    const unsigned  z = blockIdx.y;   // Globaler z-Index 
    
    const unsigned nx = blockDim.x;
    const unsigned ny = gridDim.x;
    
    const unsigned k = nx*(ny*z + y) + x;
    //////////////////////////////////////////////////////////////////////////

    if(k<size_Mat)
    {
        ////////////////////////////////////////////////////////////////////////////////
        unsigned int BC;
        BC = geoD[k];

        if( BC != GEO_SOLID &&  BC != GEO_VOID)
        {
            Distributions27 D;
            if (EvenOrOdd==true)
            {
                D.f[E   ] = &DD[E   *size_Mat];
                D.f[W   ] = &DD[W   *size_Mat];
                D.f[N   ] = &DD[N   *size_Mat];
                D.f[S   ] = &DD[S   *size_Mat];
                D.f[T   ] = &DD[T   *size_Mat];
                D.f[B   ] = &DD[B   *size_Mat];
                D.f[NE  ] = &DD[NE  *size_Mat];
                D.f[SW  ] = &DD[SW  *size_Mat];
                D.f[SE  ] = &DD[SE  *size_Mat];
                D.f[NW  ] = &DD[NW  *size_Mat];
                D.f[TE  ] = &DD[TE  *size_Mat];
                D.f[BW  ] = &DD[BW  *size_Mat];
                D.f[BE  ] = &DD[BE  *size_Mat];
                D.f[TW  ] = &DD[TW  *size_Mat];
                D.f[TN  ] = &DD[TN  *size_Mat];
                D.f[BS  ] = &DD[BS  *size_Mat];
                D.f[BN  ] = &DD[BN  *size_Mat];
                D.f[TS  ] = &DD[TS  *size_Mat];
                D.f[dirREST] = &DD[dirREST*size_Mat];
                D.f[TNE ] = &DD[TNE *size_Mat];
                D.f[TSW ] = &DD[TSW *size_Mat];
                D.f[TSE ] = &DD[TSE *size_Mat];
                D.f[TNW ] = &DD[TNW *size_Mat];
                D.f[BNE ] = &DD[BNE *size_Mat];
                D.f[BSW ] = &DD[BSW *size_Mat];
                D.f[BSE ] = &DD[BSE *size_Mat];
                D.f[BNW ] = &DD[BNW *size_Mat];
            }
            else
            {
                D.f[W   ] = &DD[E   *size_Mat];
                D.f[E   ] = &DD[W   *size_Mat];
                D.f[S   ] = &DD[N   *size_Mat];
                D.f[N   ] = &DD[S   *size_Mat];
                D.f[B   ] = &DD[T   *size_Mat];
                D.f[T   ] = &DD[B   *size_Mat];
                D.f[SW  ] = &DD[NE  *size_Mat];
                D.f[NE  ] = &DD[SW  *size_Mat];
                D.f[NW  ] = &DD[SE  *size_Mat];
                D.f[SE  ] = &DD[NW  *size_Mat];
                D.f[BW  ] = &DD[TE  *size_Mat];
                D.f[TE  ] = &DD[BW  *size_Mat];
                D.f[TW  ] = &DD[BE  *size_Mat];
                D.f[BE  ] = &DD[TW  *size_Mat];
                D.f[BS  ] = &DD[TN  *size_Mat];
                D.f[TN  ] = &DD[BS  *size_Mat];
                D.f[TS  ] = &DD[BN  *size_Mat];
                D.f[BN  ] = &DD[TS  *size_Mat];
                D.f[dirREST] = &DD[dirREST*size_Mat];
                D.f[BSW ] = &DD[TNE *size_Mat];
                D.f[BNE ] = &DD[TSW *size_Mat];
                D.f[BNW ] = &DD[TSE *size_Mat];
                D.f[BSE ] = &DD[TNW *size_Mat];
                D.f[TSW ] = &DD[BNE *size_Mat];
                D.f[TNE ] = &DD[BSW *size_Mat];
                D.f[TNW ] = &DD[BSE *size_Mat];
                D.f[TSE ] = &DD[BNW *size_Mat];
            }
            //////////////////////////////////////////////////////////////////////////
            real drho = rho[k];//0.0f;//
            real  vx1 = ux[k]; //0.0f;//
            real  vx2 = uy[k]; //0.0f;//
            real  vx3 = uz[k]; //0.0f;//
            //////////////////////////////////////////////////////////////////////////
            //index
            //////////////////////////////////////////////////////////////////////////
            unsigned int kzero= k;
            unsigned int ke   = k;
            unsigned int kw   = neighborX[k];
            unsigned int kn   = k;
            unsigned int ks   = neighborY[k];
            unsigned int kt   = k;
            unsigned int kb   = neighborZ[k];
            unsigned int ksw  = neighborY[kw];
            unsigned int kne  = k;
            unsigned int kse  = ks;
            unsigned int knw  = kw;
            unsigned int kbw  = neighborZ[kw];
            unsigned int kte  = k;
            unsigned int kbe  = kb;
            unsigned int ktw  = kw;
            unsigned int kbs  = neighborZ[ks];
            unsigned int ktn  = k;
            unsigned int kbn  = kb;
            unsigned int kts  = ks;
            unsigned int ktse = ks;
            unsigned int kbnw = kbw;
            unsigned int ktnw = kw;
            unsigned int kbse = kbs;
            unsigned int ktsw = ksw;
            unsigned int kbne = kb;
            unsigned int ktne = k;
            unsigned int kbsw = neighborZ[ksw];
	        //////////////////////////////////////////////////////////////////////////////
	        //neighbor index
	        uint kPx   = neighborX[k];
	        uint kPy   = neighborY[k];
	        uint kPz   = neighborZ[k];
	        uint kMxyz = neighborWSB[k];
	        uint kMx   = neighborZ[neighborY[kMxyz]];
	        uint kMy   = neighborZ[neighborX[kMxyz]];
	        uint kMz   = neighborY[neighborX[kMxyz]];
            //////////////////////////////////////////////////////////////////////////
	        //getVeloX//
	        real vx1NeighborPx = ux[kPx];
	        real vx1NeighborMx = ux[kMx];
	        real vx1NeighborPy = ux[kPy];
	        real vx1NeighborMy = ux[kMy];
	        real vx1NeighborPz = ux[kPz];
	        real vx1NeighborMz = ux[kMz];
	        //getVeloY//
	        real vx2NeighborPx = uy[kPx];
	        real vx2NeighborMx = uy[kMx];
	        real vx2NeighborPy = uy[kPy];
	        real vx2NeighborMy = uy[kMy];
	        real vx2NeighborPz = uy[kPz];
	        real vx2NeighborMz = uy[kMz];
	        //getVeloZ//
	        real vx3NeighborPx = uz[kPx];
	        real vx3NeighborMx = uz[kMx];
	        real vx3NeighborPy = uz[kPy];
	        real vx3NeighborMy = uz[kMy];
	        real vx3NeighborPz = uz[kPz];
	        real vx3NeighborMz = uz[kMz];
            //////////////////////////////////////////////////////////////////////////

	        real dvx1dx = (vx1NeighborPx - vx1NeighborMx) / c2o1;
	        real dvx1dy = (vx1NeighborPy - vx1NeighborMy) / c2o1;
	        real dvx1dz = (vx1NeighborPz - vx1NeighborMz) / c2o1;

	        real dvx2dx = (vx2NeighborPx - vx2NeighborMx) / c2o1;
	        real dvx2dy = (vx2NeighborPy - vx2NeighborMy) / c2o1;
	        real dvx2dz = (vx2NeighborPz - vx2NeighborMz) / c2o1;

	        real dvx3dx = (vx3NeighborPx - vx3NeighborMx) / c2o1;
	        real dvx3dy = (vx3NeighborPy - vx3NeighborMy) / c2o1;
	        real dvx3dz = (vx3NeighborPz - vx3NeighborMz) / c2o1;

            //////////////////////////////////////////////////////////////////////////

            // the following code is copy and pasted from VirtualFluidsCore/Visitors/InitDistributionsBlockVisitor.cpp
            // i.e. Konstantins code

            real ax = dvx1dx;
            real ay = dvx1dy;
            real az = dvx1dz;

            real bx = dvx2dx;
            real by = dvx2dy;
            real bz = dvx2dz;

            real cx = dvx3dx;
            real cy = dvx3dy;
            real cz = dvx3dz;

            real eps_new = c1o1;
            real op      = c1o1;
            real o       = omega;

            real f_E    =            eps_new *((5.*ax*o + 5.*by*o + 5.*cz*o - 8.*ax*op + 4.*by*op + 4.*cz*op)/(54.*o*op));

            real f_N    =    f_E   + eps_new *((2.*(ax - by))/(9.*o));
            real f_T    =    f_E   + eps_new *((2.*(ax - cz))/(9.*o));
            real f_NE   =            eps_new *(-(5.*cz*o + 3.*(ay + bx)*op - 2.*cz*op + ax*(5.*o + op) + by*(5.*o + op))/(54.*o*op));
            real f_SE   =    f_NE  + eps_new *((  ay + bx )/(9.*o));
            real f_TE   =            eps_new *(-(5.*cz*o + by*(5.*o - 2.*op) + 3.*(az + cx)*op + cz*op + ax*(5.*o + op))/(54.*o*op));
            real f_BE   =    f_TE  + eps_new *((  az + cx )/(9.*o));
            real f_TN   =            eps_new *(-(5.*ax*o + 5.*by*o + 5.*cz*o - 2.*ax*op + by*op + 3.*bz*op + 3.*cy*op + cz*op)/(54.*o*op));
            real f_BN   =    f_TN  + eps_new *((  bz + cy )/(9.*o));
            real f_ZERO =            eps_new *((5.*(ax + by + cz))/(9.*op));
            real f_TNE  =            eps_new *(-(ay + az + bx + bz + cx + cy)/(72.*o));
            real f_TSW  =  - f_TNE - eps_new *((ay + bx)/(36.*o));
            real f_TSE  =  - f_TNE - eps_new *((az + cx)/(36.*o));
            real f_TNW  =  - f_TNE - eps_new *((bz + cy)/(36.*o));

            //////////////////////////////////////////////////////////////////////////
            real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

            (D.f[dirREST])[kzero] =   c8o27* (drho-cu_sq*(c1o1+drho));
            (D.f[E   ])[ke   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq));
            (D.f[W   ])[kw   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq));
            (D.f[N   ])[kn   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq));
            (D.f[S   ])[ks   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
            (D.f[T   ])[kt   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq));
            (D.f[B   ])[kb   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq));
            (D.f[NE  ])[kne  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
            (D.f[SW  ])[ksw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
            (D.f[SE  ])[kse  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
            (D.f[NW  ])[knw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
            (D.f[TE  ])[kte  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
            (D.f[BW  ])[kbw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
            (D.f[BE  ])[kbe  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
            (D.f[TW  ])[ktw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
            (D.f[TN  ])[ktn  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
            (D.f[BS  ])[kbs  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
            (D.f[BN  ])[kbn  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
            (D.f[TS  ])[kts  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
            (D.f[TNE ])[ktne ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
            (D.f[BSW ])[kbsw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
            (D.f[BNE ])[kbne ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
            (D.f[TSW ])[ktsw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
            (D.f[TSE ])[ktse ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
            (D.f[BNW ])[kbnw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
            (D.f[BSE ])[kbse ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
            (D.f[TNW ])[ktnw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));

            //////////////////////////////////////////////////////////////////////////

            (D.f[dirREST])[kzero] += (c1o1+drho) * f_ZERO;
            (D.f[E   ])[ke   ] += (c1o1+drho) * f_E   ;
            (D.f[W   ])[kw   ] += (c1o1+drho) * f_E   ;
            (D.f[N   ])[kn   ] += (c1o1+drho) * f_N   ;
            (D.f[S   ])[ks   ] += (c1o1+drho) * f_N   ;
            (D.f[T   ])[kt   ] += (c1o1+drho) * f_T   ;
            (D.f[B   ])[kb   ] += (c1o1+drho) * f_T   ;
            (D.f[NE  ])[kne  ] += (c1o1+drho) * f_NE  ;
            (D.f[SW  ])[ksw  ] += (c1o1+drho) * f_NE  ;
            (D.f[SE  ])[kse  ] += (c1o1+drho) * f_SE  ;
            (D.f[NW  ])[knw  ] += (c1o1+drho) * f_SE  ;
            (D.f[TE  ])[kte  ] += (c1o1+drho) * f_TE  ;
            (D.f[BW  ])[kbw  ] += (c1o1+drho) * f_TE  ;
            (D.f[BE  ])[kbe  ] += (c1o1+drho) * f_BE  ;
            (D.f[TW  ])[ktw  ] += (c1o1+drho) * f_BE  ;
            (D.f[TN  ])[ktn  ] += (c1o1+drho) * f_TN  ;
            (D.f[BS  ])[kbs  ] += (c1o1+drho) * f_TN  ;
            (D.f[BN  ])[kbn  ] += (c1o1+drho) * f_BN  ;
            (D.f[TS  ])[kts  ] += (c1o1+drho) * f_BN  ;
            (D.f[TNE ])[ktne ] += (c1o1+drho) * f_TNE ;
            (D.f[BSW ])[kbsw ] += (c1o1+drho) * f_TNE ;
            (D.f[BNE ])[kbne ] += (c1o1+drho) * f_TSW ;
            (D.f[TSW ])[ktsw ] += (c1o1+drho) * f_TSW ;
            (D.f[TSE ])[ktse ] += (c1o1+drho) * f_TSE ;
            (D.f[BNW ])[kbnw ] += (c1o1+drho) * f_TSE ;
            (D.f[BSE ])[kbse ] += (c1o1+drho) * f_TNW ;
            (D.f[TNW ])[ktnw ] += (c1o1+drho) * f_TNW ;

            //////////////////////////////////////////////////////////////////////////
        }
	    else
	    {
		    //////////////////////////////////////////////////////////////////////////
		    Distributions27 D;
		    D.f[dirREST] = &DD[dirREST*size_Mat];
		    //////////////////////////////////////////////////////////////////////////
		    (D.f[dirREST])[k] = c96o1;
		    //////////////////////////////////////////////////////////////////////////
	    }
   }
}