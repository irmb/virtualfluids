#include "hip/hip_runtime.h"
#include "ActuatorLine.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Kernel/Utilities/CudaGrid.h"
#include "lbm/constants/NumericConstants.h"
#include "VirtualFluids_GPU/GPU/GeometryUtils.h"

__host__ __device__ __inline__ real calcGaussian3D(real posX, real posY, real posZ, real destX, real destY, real destZ, real epsilon)
{
    real distX = destX-posX;
    real distY = destY-posY;
    real distZ = destZ-posZ;
    real dist = sqrt(distX*distX+distY*distY+distZ*distZ);
    return pow(epsilon,-3)*pow(vf::lbm::constant::cPi,-1.5f)*exp(-pow(dist/epsilon,2));
}


__global__ void interpolateVelocities(real* gridCoordsX, real* gridCoordsY, real* gridCoordsZ, 
                                      uint* neighborsX, uint* neighborsY, uint* neighborsZ, 
                                      uint* neighborsWSB, 
                                      real* vx, real* vy, real* vz, 
                                      int numberOfIndices, 
                                      real* bladeCoordsX, real* bladeCoordsY, real* bladeCoordsZ, 
                                      real* bladeVelocitiesX, real* bladeVelocitiesY, real* bladeVelocitiesZ, 
                                      uint* bladeIndices, int numberOfNodes)
{
    // Possibly restructure this to check every cell whether it is bsw of some blade node and then do interpolation
    // --> no need to save closest nodes and search new closest node
    const uint x = threadIdx.x; 
    const uint y = blockIdx.x;
    const uint z = blockIdx.y;

    const uint nx = blockDim.x;
    const uint ny = gridDim.x;

    const uint node = nx*(ny*z + y) + x;

    if(node>=numberOfNodes) return;

    real bladePosX = bladeCoordsX[node];
    real bladePosY = bladeCoordsY[node];
    real bladePosZ = bladeCoordsZ[node];

    uint old_index = bladeIndices[node];
    // if(node==0 or node==90)
    // {
    //     printf("before: blade (%f, %f, %f), node BSW (%f, %f, %f), nodeTNE (%f, %f, %f)\n", bladePosX, bladePosY, bladePosZ, gridCoordsX[old_index], gridCoordsY[old_index], gridCoordsZ[old_index], gridCoordsX[neighborsX[old_index]], gridCoordsY[neighborsY[old_index]], gridCoordsZ[neighborsZ[old_index]]);
    // }
    uint k, ke, kn, kt;
    uint kne, kte, ktn, ktne;

    k = findNearestCellBSW(old_index, 
                           gridCoordsX, gridCoordsY, gridCoordsZ, 
                           bladePosX, bladePosY, bladePosZ, 
                           neighborsX, neighborsY, neighborsZ, neighborsWSB);
        
    bladeIndices[node] = k;

    getNeighborIndicesBSW(k, ke, kn, kt, kne, kte, ktn, ktne, neighborsX, neighborsY, neighborsZ);

    real dW, dE, dN, dS, dT, dB;

    real invDeltaX = 1.f/(gridCoordsX[ktne]-gridCoordsX[k]);
    real distX = invDeltaX*(gridCoordsX[ktne]-bladePosX);
    real distY = invDeltaX*(gridCoordsY[ktne]-bladePosY);
    real distZ = invDeltaX*(gridCoordsZ[ktne]-bladePosZ);

    getInterpolationWeights(dW, dE, dN, dS, dT, dB, 
                            distX, distY, distZ);

    bladeVelocitiesX[node] = trilinearInterpolation(dW, dE, dN, dS, dT, dB, k, ke, kn, kt, kne, kte, ktn, ktne, vx);
    bladeVelocitiesY[node] = trilinearInterpolation(dW, dE, dN, dS, dT, dB, k, ke, kn, kt, kne, kte, ktn, ktne, vy);
    bladeVelocitiesZ[node] = trilinearInterpolation(dW, dE, dN, dS, dT, dB, k, ke, kn, kt, kne, kte, ktn, ktne, vz);

    // if(node==numberOfNodes-1)
    // {
    //     printf("after: blade (%f, %f, %f), node BSW (%f, %f, %f), nodeTNE (%f, %f, %f)\n", bladePosX, bladePosY, bladePosZ, gridCoordsX[kBSW], gridCoordsY[kBSW], gridCoordsZ[kBSW], gridCoordsX[neighborsX[kBSW]], gridCoordsY[neighborsY[kBSW]], gridCoordsZ[neighborsZ[kBSW]]);
    // }

}


__global__ void applyBodyForces(real* gridCoordsX, real* gridCoordsY, real* gridCoordsZ,
                           real* gridForcesX, real* gridForcesY, real* gridForcesZ, 
                           uint* gridIndices, int numberOfIndices, 
                           real* bladeCoordsX, real* bladeCoordsY, real* bladeCoordsZ, 
                           real* bladeForcesX, real* bladeForcesY,real* bladeForcesZ,
                           real* bladeRadii,
                           real radius,
                           int nBlades, int nBladeNodes,
                           real epsilon, real delta_x)
{
    const uint x = threadIdx.x; 
    const uint y = blockIdx.x;
    const uint z = blockIdx.y;

    const uint nx = blockDim.x;
    const uint ny = gridDim.x;

    const uint index = nx*(ny*z + y) + x;

    if(index>=numberOfIndices) return;

    int gridIndex = gridIndices[index];

    real posX = gridCoordsX[gridIndex];
    real posY = gridCoordsY[gridIndex];
    real posZ = gridCoordsZ[gridIndex];

    real fXYZ_X = 0.0f;
    real fXYZ_Y = 0.0f;
    real fXYZ_Z = 0.0f;

    real eta = 0.0f;

    real delta_x_cubed = pow(delta_x,3);

    for( uint blade=0; blade<nBlades; blade++)
    {    
        real last_r = 0.0f;
        real r = 0.0f;

        for( uint bladeNode=0; bladeNode<nBladeNodes; bladeNode++)
        {
            int node = bladeNode+blade*nBladeNodes;
            eta = calcGaussian3D(posX, posY, posZ, bladeCoordsX[node], bladeCoordsY[node], bladeCoordsZ[node], epsilon)*delta_x_cubed;
            r = bladeRadii[bladeNode];

            fXYZ_X += bladeForcesX[node]*(r-last_r)*eta;
            fXYZ_Y += bladeForcesY[node]*(r-last_r)*eta;
            fXYZ_Z += bladeForcesZ[node]*(r-last_r)*eta;

            last_r = r;

            // if(node==16||node==48||node==80)
            // {            
                // printf("uRTZ: %f %f %f \n", uRTZ_X, uRTZ_Y, uRTZ_Z);
                // printf("uXYZ: %f %f %f \n", uXYZ_X, uXYZ_Y, uXYZ_Z);
                // printf("omega: %f radius: %f \n", this->omega, r);
                // printf("force ratio %f \n", forceRatio);
                // printf("u_rel: %f v_rel: %f \n", u_rel, v_rel);
                // printf("c: %f, cn: %f ct: %f \n", c, Cn, Ct);
                // printf("fXYZ: %f %f %f \n", fXYZ_X, fXYZ_Y, fXYZ_Z);
                // printf("fRTZ: %f %f %f \n", fRTZ_X, fRTZ_Y, fRTZ_Z);
                // printf("X Y Z: %f %f %f \n", this->bladeCoordsXH[node],this->bladeCoordsYH[node],this->bladeCoordsZH[node]);
            // }
        }    

        fXYZ_X += bladeForcesX[nBladeNodes-1]*(radius-last_r)*eta;
        fXYZ_Y += bladeForcesY[nBladeNodes-1]*(radius-last_r)*eta;
        fXYZ_Z += bladeForcesZ[nBladeNodes-1]*(radius-last_r)*eta;
    }

    gridForcesX[gridIndex] = fXYZ_X;
    gridForcesY[gridIndex] = fXYZ_Y;
    gridForcesZ[gridIndex] = fXYZ_Z;
}


void ActuatorLine::init(Parameter* para, GridProvider* gridProvider, CudaMemoryManager* cudaManager)
{
    this->initBladeRadii(cudaManager);
    this->initBladeCoords(cudaManager);    
    this->initBladeIndices(para, cudaManager);
    this->initBladeVelocities(cudaManager);
    this->initBladeForces(cudaManager);    
    this->initBoundingSphere(para, cudaManager);
}


void ActuatorLine::visit(Parameter* para, CudaMemoryManager* cudaManager, int level, unsigned int t)
{
    if (level != this->level) return;
    
    cudaManager->cudaCopyBladeCoordsHtoD(this);

    unsigned int numberOfThreads = 128;
    vf::gpu::CudaGrid bladeGrid = vf::gpu::CudaGrid(numberOfThreads, this->numberOfNodes);

    interpolateVelocities<<< bladeGrid.grid, bladeGrid.threads >>>(
        para->getParD(this->level)->coordX_SP, para->getParD(this->level)->coordY_SP, para->getParD(this->level)->coordZ_SP,        
        para->getParD(this->level)->neighborX_SP, para->getParD(this->level)->neighborY_SP, para->getParD(this->level)->neighborZ_SP, para->getParD(this->level)->neighborWSB_SP,
        para->getParD(this->level)->vx_SP, para->getParD(this->level)->vy_SP, para->getParD(this->level)->vz_SP,
        this->numberOfIndices,
        this->bladeCoordsXD, this->bladeCoordsYD, this->bladeCoordsZD,  
        this->bladeVelocitiesXD, this->bladeVelocitiesYD, this->bladeVelocitiesZD,  
        this->bladeIndicesD, this->numberOfNodes);

    cudaManager->cudaCopyBladeVelocitiesDtoH(this);

    if(true)
    {
        this->calcForcesEllipticWing(para);
    }

    cudaManager->cudaCopyBladeForcesHtoD(this);

    vf::gpu::CudaGrid sphereGrid = vf::gpu::CudaGrid(numberOfThreads, this->numberOfIndices);

    applyBodyForces<<<sphereGrid.grid, sphereGrid.threads>>>(
        para->getParD(this->level)->coordX_SP, para->getParD(this->level)->coordY_SP, para->getParD(this->level)->coordZ_SP,        
        para->getParD(this->level)->forceX_SP, para->getParD(this->level)->forceY_SP, para->getParD(this->level)->forceZ_SP,        
        this->boundingSphereIndicesD, this->numberOfIndices,
        this->bladeCoordsXD, this->bladeCoordsYD, this->bladeCoordsZD,  
        this->bladeForcesXD, this->bladeForcesYD, this->bladeForcesZD,
        this->bladeRadiiD,
        this->diameter*0.5f,  
        this->nBlades, this->nBladeNodes,
        this->epsilon, this->delta_x);

    real dazimuth = this->omega*this->delta_t;

    this->azimuth += dazimuth;
    this->rotateBlades(dazimuth);
}


void ActuatorLine::free(Parameter* para, CudaMemoryManager* cudaManager)
{
    cudaManager->cudaFreeBladeRadii(this);
    cudaManager->cudaFreeBladeCoords(this);
    cudaManager->cudaFreeBladeVelocities(this);
    cudaManager->cudaFreeBladeForces(this);
    cudaManager->cudaFreeBladeIndices(this);

    cudaManager->cudaFreeSphereIndices(this);
}


void ActuatorLine::calcForcesEllipticWing(Parameter* para)
{
    real localAzimuth;
    uint node;
    real uXYZ_X, uXYZ_Y, uXYZ_Z;
    real uRTZ_X, uRTZ_Y, uRTZ_Z;
    real fXYZ_X, fXYZ_Y, fXYZ_Z;
    real fRTZ_X, fRTZ_Y, fRTZ_Z;
    real r;
    real u_rel, v_rel, u_rel_sq;
    real phi;
    real Cl = 1.f;
    real Cd = 0.f;
    real c0 = 1.f;

    real c, Cn, Ct;

    real forceRatio = this->density*pow(this->delta_x,4)*pow(this->delta_t,-2);

    for( int blade=0; blade<this->nBlades; blade++)
    {
        localAzimuth = this->azimuth+2*blade*vf::lbm::constant::cPi/this->nBlades;
        for( uint bladeNode=0; bladeNode<this->nBladeNodes; bladeNode++)
        {
            node = bladeNode+blade*this->nBladeNodes;
            uXYZ_X = this->bladeVelocitiesXH[node]*para->getVelocityRatio();
            uXYZ_Y = this->bladeVelocitiesYH[node]*para->getVelocityRatio();
            uXYZ_Z = this->bladeVelocitiesZH[node]*para->getVelocityRatio();

            invRotateAboutX3D(localAzimuth, uXYZ_X, uXYZ_Y, uXYZ_Z, uRTZ_X, uRTZ_Y, uRTZ_Z);
            r = this->bladeRadiiH[bladeNode];

            u_rel = uRTZ_X;
            v_rel = uRTZ_Y+this->omega*r;
            u_rel_sq = u_rel*u_rel+v_rel*v_rel;
            phi = atan2(u_rel, v_rel);

            c = c0 * sqrt( 1.f- pow(4.f*r/this->diameter-1.f, 2.f) );
            Cn =   Cl*cos(phi)+Cd*sin(phi);
            Ct =  -Cl*sin(phi)+Cd*cos(phi);

            fRTZ_X = 0.5f*u_rel_sq*c*this->density*Cn;
            fRTZ_Y = 0.5f*u_rel_sq*c*this->density*Ct;
            fRTZ_Z = 0.0;

            rotateAboutX3D(localAzimuth, fRTZ_X, fRTZ_Y, fRTZ_Z, fXYZ_X, fXYZ_Y, fXYZ_Z);
        
            this->bladeForcesXH[node] = fXYZ_X/forceRatio;
            this->bladeForcesYH[node] = fXYZ_Y/forceRatio;
            this->bladeForcesZH[node] = fXYZ_Z/forceRatio;

            // if(node==16||node==48||node==80)
            // {            
            // printf("uRTZ: %f %f %f \n", uRTZ_X, uRTZ_Y, uRTZ_Z);
            // printf("uXYZ: %f %f %f \n", uXYZ_X, uXYZ_Y, uXYZ_Z);
            // printf("omega: %f radius: %f \n", this->omega, r);
            // printf("force ratio %f \n", forceRatio);
            // printf("u_rel: %f v_rel: %f \n", u_rel, v_rel);
            // printf("c: %f, cn: %f ct: %f \n", c, Cn, Ct);
            // printf("fXYZ: %f %f %f \n", fXYZ_X, fXYZ_Y, fXYZ_Z);
            // printf("fRTZ: %f %f %f \n", fRTZ_X, fRTZ_Y, fRTZ_Z);
            // printf("X Y Z: %f %f %f \n", this->bladeCoordsXH[node],this->bladeCoordsYH[node],this->bladeCoordsZH[node]);
            // }
        }
    }
    // printf("uRTZ: %f %f %f \n", uRTZ_X, uRTZ_Y, uRTZ_Z);
    // printf("uXYZ: %f %f %f \n", uXYZ_X, uXYZ_Y, uXYZ_Z);
    // printf("omega: %f radius: %f \n", this->omega, r);

    // printf("u_rel: %f v_rel: %f \n", u_rel, v_rel);
    // printf("c: %f, cn: %f ct: %f \n", c, Cn, Ct);
    // printf("fXYZ: %f %f %f \n", fXYZ_X, fXYZ_Y, fXYZ_Z);
    // printf("fRTZ: %f %f %f \n", fRTZ_X, fRTZ_Y, fRTZ_Z);

}

void ActuatorLine::rotateBlades(real angle)
{
    for(unsigned int node=0; node<this->nBladeNodes*this->nBlades; node++)
    {
        real oldCoordX = this->bladeCoordsXH[node];
        real oldCoordY = this->bladeCoordsYH[node];
        real oldCoordZ = this->bladeCoordsZH[node];

        real newCoordX, newCoordY, newCoordZ;
        rotateAboutX3D(angle, oldCoordX, oldCoordY, oldCoordZ, newCoordX, newCoordY, newCoordZ, this->turbinePosX, this->turbinePosY, this->turbinePosZ);
        
        this->bladeCoordsYH[node] = newCoordX;
        this->bladeCoordsYH[node] = newCoordY;
        this->bladeCoordsZH[node] = newCoordZ;
    }
}

void ActuatorLine::initBladeRadii(CudaMemoryManager* cudaManager)
{   
    cudaManager->cudaAllocBladeRadii(this);

    real dx = 0.5f*this->diameter/this->nBladeNodes;        
    for(uint node=0; node<this->nBladeNodes; node++)
    {
        this->bladeRadiiH[node] = dx*(node+1);
    }
    cudaManager->cudaCopyBladeRadiiHtoD(this);
}

void ActuatorLine::initBladeCoords(CudaMemoryManager* cudaManager)
{   
    cudaManager->cudaAllocBladeCoords(this);

    for( unsigned int blade=0; blade<this->nBlades; blade++)
    {
        real localAzimuth = this->azimuth+(2*vf::lbm::constant::cPi/this->nBlades)*blade;
        for(unsigned int node=0; node<this->nBladeNodes; node++)
        {
            real coordX, coordY, coordZ;
            real x,y,z;
            x = 0.f;
            y = 0.f;
            z = this->bladeRadiiH[node];
            rotateAboutX3D(localAzimuth, x, y, z, coordX, coordY, coordZ);
            this->bladeCoordsXH[node+this->nBladeNodes*blade] = coordX+this->turbinePosX;
            this->bladeCoordsYH[node+this->nBladeNodes*blade] = coordY+this->turbinePosY;
            this->bladeCoordsZH[node+this->nBladeNodes*blade] = coordZ+this->turbinePosZ;
            // printf("blade: %i, az %f , x %f, y %f , z %f \n", blade, localAzimuth, coordX, coordY, coordZ);
        }
    }
    cudaManager->cudaCopyBladeCoordsHtoD(this);
}

void ActuatorLine::initBladeVelocities(CudaMemoryManager* cudaManager)
{   
    cudaManager->cudaAllocBladeVelocities(this);

    for(unsigned int node=0; node<this->numberOfNodes; node++)
    {
        this->bladeVelocitiesXH[node] = 0.f;
        this->bladeVelocitiesYH[node] = 0.f;
        this->bladeVelocitiesZH[node] = 0.f;
    }
    cudaManager->cudaCopyBladeVelocitiesHtoD(this);
}

void ActuatorLine::initBladeForces(CudaMemoryManager* cudaManager)
{   
    cudaManager->cudaAllocBladeForces(this);

    for(unsigned int node=0; node<this->numberOfNodes; node++)
    {
        this->bladeForcesXH[node] = 0.f;
        this->bladeForcesYH[node] = 0.f;
        this->bladeForcesZH[node] = 0.f;
    }
    cudaManager->cudaCopyBladeForcesHtoD(this);
}

void ActuatorLine::initBladeIndices(Parameter* para, CudaMemoryManager* cudaManager)
{   
    cudaManager->cudaAllocBladeIndices(this);

    real* coordsX = para->getParH(this->level)->coordX_SP;
    real* coordsY = para->getParH(this->level)->coordY_SP;
    real* coordsZ = para->getParH(this->level)->coordZ_SP;

    for(unsigned int node=0; node<this->numberOfNodes; node++)
    {
        this->bladeIndicesH[node] = findNearestCellBSW(1, coordsX, coordsY, coordsZ, 
                                                       this->bladeCoordsXH[node], this->bladeCoordsYH[node], this->bladeCoordsZH[node],
                                                       para->getParH(this->level)->neighborX_SP, para->getParH(this->level)->neighborY_SP, para->getParH(this->level)->neighborZ_SP,
                                                       para->getParH(this->level)->neighborWSB_SP);
        
    }
    cudaManager->cudaCopyBladeIndicesHtoD(this);
}

void ActuatorLine::initBoundingSphere(Parameter* para, CudaMemoryManager* cudaManager)
{
    // Actuator line exists only on 1 level
    std::vector<int> nodesInSphere;

    for (int j = 1; j <= para->getParH(this->level)->size_Mat_SP; j++)
    {
        const real coordX = para->getParH(this->level)->coordX_SP[j];
        const real coordY = para->getParH(this->level)->coordY_SP[j];
        const real coordZ = para->getParH(this->level)->coordZ_SP[j];
        const real dist = sqrt(pow(coordX-this->turbinePosX,2)+pow(coordY-this->turbinePosY,2)+pow(coordZ-this->turbinePosZ,2));
        
        if(dist < 0.6*this->diameter) nodesInSphere.push_back(j);
    }

    this->numberOfIndices = nodesInSphere.size();
    cudaManager->cudaAllocSphereIndices(this);
    std::copy(nodesInSphere.begin(), nodesInSphere.end(), this->boundingSphereIndicesH);
    cudaManager->cudaCopySphereIndicesHtoD(this);
}