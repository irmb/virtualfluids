#include "hip/hip_runtime.h"
#include "Probe.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


void Probe::init(Parameter* para, GridProvider* gridProvider, CudaMemoryManager* cudaManager)
{

    probeParams.resize(para->getMaxLevel()+1);

    for(int level=0; level<=para->getMaxLevel(); level++)
    {
        std::vector<int> probeIndices_level;
        std::vector<real> distX_level;
        std::vector<real> distY_level;
        std::vector<real> distZ_level;
        real dx = abs(para->getParH(level)->coordX_SP[1]-para->getParH(level)->coordX_SP[para->getParH(level)->neighborX_SP[1]]);
        for(uint j=0; j<para->getParH(level)->size_Mat_SP; j++ )
        {    
            for(uint point=0; point<this->nProbePoints; point++)
            {
                real distX = this->pointCoordsX[point]-para->getParH(level)->coordX_SP[j];
                real distY = this->pointCoordsY[point]-para->getParH(level)->coordY_SP[j];
                real distZ = this->pointCoordsZ[point]-para->getParH(level)->coordZ_SP[j];
                if( distX <=dx && distY <=dx && distZ <=dx &&
                    distX >0.f && distY >0.f && distZ >0.f)
                {
                    probeIndices_level.push_back(j);
                    distX_level.push_back( distX/dx );
                    distY_level.push_back( distY/dx );
                    distZ_level.push_back( distZ/dx );
                    // printf("Found Point %i, x: %f, y: %f, z: %f, \n For %f %f %f, \n distx: %f, disty: %f, distz: %f \n", j, para->getParH(level)->coordX_SP[j],para->getParH(level)->coordY_SP[j],para->getParH(level)->coordZ_SP[j],
                    // this->pointCoordsX[point], this->pointCoordsY[point], this->pointCoordsZ[point], 
                    // distX, distY, distZ);
                }
            }
        }

        probeParams[level] = new ProbeStruct;
        probeParams[level]->nPoints = probeIndices_level.size();
        // Might have to catch nPoints=0 ?!?!
        checkCudaErrors( hipHostMalloc((void**) &probeParams[level]->distXH,        sizeof(real)*probeParams[level]->nPoints) );
        checkCudaErrors( hipHostMalloc((void**) &probeParams[level]->distYH,        sizeof(real)*probeParams[level]->nPoints) );
        checkCudaErrors( hipHostMalloc((void**) &probeParams[level]->distZH,        sizeof(real)*probeParams[level]->nPoints) );
        checkCudaErrors( hipHostMalloc((void**) &probeParams[level]->pointIndicesH, sizeof(int)*probeParams[level]->nPoints) );

        checkCudaErrors( hipMalloc    ((void**) &probeParams[level]->distXD,        sizeof(real)*probeParams[level]->nPoints) );
        checkCudaErrors( hipMalloc    ((void**) &probeParams[level]->distYD,        sizeof(real)*probeParams[level]->nPoints) );
        checkCudaErrors( hipMalloc    ((void**) &probeParams[level]->distZD,        sizeof(real)*probeParams[level]->nPoints) );
        checkCudaErrors( hipMalloc    ((void**) &probeParams[level]->pointIndicesD, sizeof(int)*probeParams[level]->nPoints) );

        std::copy(distX_level.begin(), distX_level.end(), probeParams[level]->distXH);
        std::copy(distY_level.begin(), distY_level.end(), probeParams[level]->distYH);
        std::copy(distZ_level.begin(), distZ_level.end(), probeParams[level]->distZH);
        std::copy(probeIndices_level.begin(), probeIndices_level.end(), probeParams[level]->pointIndicesH);
    }
}


void Probe::visit(Parameter* para, int level, unsigned int t)
{

}

void Probe::setProbePointsFromList(std::vector<real> &_pointCoordsX, std::vector<real> &_pointCoordsY, std::vector<real> &_pointCoordsZ)
{
    bool isSameLength = ( (_pointCoordsX.size()==_pointCoordsY.size()) && (_pointCoordsY.size()==_pointCoordsZ.size()));
    assert("Probe: point lists have different lengths" && isSameLength);
    this->pointCoordsX = _pointCoordsX;
    this->pointCoordsY = _pointCoordsY;
    this->pointCoordsZ = _pointCoordsZ;
    this->nProbePoints = _pointCoordsX.size();
    printf("Adde list of %u  points", this->nProbePoints );
}