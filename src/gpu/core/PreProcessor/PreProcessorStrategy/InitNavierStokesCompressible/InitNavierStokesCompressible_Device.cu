#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \author Martin Schoenherr
//=======================================================================================
#include "Calculation/Calculation.h" 
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;
#include "math.h"

#include <stdio.h>

__global__ void InitNavierStokesCompressible_Device(
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned int* geoD,
    real* rho,
    real* ux,
    real* uy,
    real* uz,
    unsigned int size_Mat,
    real* DD,
    bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   geoD[k];

      if( BC != GEO_SOLID &&  BC != GEO_VOID)
      {
         Distributions27 D;
         if (EvenOrOdd==true)
         {
            D.f[dP00   ] = &DD[dP00   *size_Mat];
            D.f[dM00   ] = &DD[dM00   *size_Mat];
            D.f[d0P0   ] = &DD[d0P0   *size_Mat];
            D.f[d0M0   ] = &DD[d0M0   *size_Mat];
            D.f[d00P   ] = &DD[d00P   *size_Mat];
            D.f[d00M   ] = &DD[d00M   *size_Mat];
            D.f[dPP0  ] = &DD[dPP0  *size_Mat];
            D.f[dMM0  ] = &DD[dMM0  *size_Mat];
            D.f[dPM0  ] = &DD[dPM0  *size_Mat];
            D.f[dMP0  ] = &DD[dMP0  *size_Mat];
            D.f[dP0P  ] = &DD[dP0P  *size_Mat];
            D.f[dM0M  ] = &DD[dM0M  *size_Mat];
            D.f[dP0M  ] = &DD[dP0M  *size_Mat];
            D.f[dM0P  ] = &DD[dM0P  *size_Mat];
            D.f[d0PP  ] = &DD[d0PP  *size_Mat];
            D.f[d0MM  ] = &DD[d0MM  *size_Mat];
            D.f[d0PM  ] = &DD[d0PM  *size_Mat];
            D.f[d0MP  ] = &DD[d0MP  *size_Mat];
            D.f[d000] = &DD[d000*size_Mat];
            D.f[dPPP ] = &DD[dPPP *size_Mat];
            D.f[dMMP ] = &DD[dMMP *size_Mat];
            D.f[dPMP ] = &DD[dPMP *size_Mat];
            D.f[dMPP ] = &DD[dMPP *size_Mat];
            D.f[dPPM ] = &DD[dPPM *size_Mat];
            D.f[dMMM ] = &DD[dMMM *size_Mat];
            D.f[dPMM ] = &DD[dPMM *size_Mat];
            D.f[dMPM ] = &DD[dMPM *size_Mat];
         }
         else
         {
            D.f[dM00   ] = &DD[dP00   *size_Mat];
            D.f[dP00   ] = &DD[dM00   *size_Mat];
            D.f[d0M0   ] = &DD[d0P0   *size_Mat];
            D.f[d0P0   ] = &DD[d0M0   *size_Mat];
            D.f[d00M   ] = &DD[d00P   *size_Mat];
            D.f[d00P   ] = &DD[d00M   *size_Mat];
            D.f[dMM0  ] = &DD[dPP0  *size_Mat];
            D.f[dPP0  ] = &DD[dMM0  *size_Mat];
            D.f[dMP0  ] = &DD[dPM0  *size_Mat];
            D.f[dPM0  ] = &DD[dMP0  *size_Mat];
            D.f[dM0M  ] = &DD[dP0P  *size_Mat];
            D.f[dP0P  ] = &DD[dM0M  *size_Mat];
            D.f[dM0P  ] = &DD[dP0M  *size_Mat];
            D.f[dP0M  ] = &DD[dM0P  *size_Mat];
            D.f[d0MM  ] = &DD[d0PP  *size_Mat];
            D.f[d0PP  ] = &DD[d0MM  *size_Mat];
            D.f[d0MP  ] = &DD[d0PM  *size_Mat];
            D.f[d0PM  ] = &DD[d0MP  *size_Mat];
            D.f[d000] = &DD[d000*size_Mat];
            D.f[dMMM ] = &DD[dPPP *size_Mat];
            D.f[dPPM ] = &DD[dMMP *size_Mat];
            D.f[dMPM ] = &DD[dPMP *size_Mat];
            D.f[dPMM ] = &DD[dMPP *size_Mat];
            D.f[dMMP ] = &DD[dPPM *size_Mat];
            D.f[dPPP ] = &DD[dMMM *size_Mat];
            D.f[dMPP ] = &DD[dPMM *size_Mat];
            D.f[dPMP ] = &DD[dMPM *size_Mat];
         }
         //////////////////////////////////////////////////////////////////////////
         real drho = rho[k];//0.0f;//
         real  vx1 = ux[k]; //0.0f;//
         real  vx2 = uy[k]; //0.0f;//
         real  vx3 = uz[k]; //0.0f;//
         //////////////////////////////////////////////////////////////////////////
         //index
         //////////////////////////////////////////////////////////////////////////
         unsigned int kzero= k;
         unsigned int ke   = k;
         unsigned int kw   = neighborX[k];
         unsigned int kn   = k;
         unsigned int ks   = neighborY[k];
         unsigned int kt   = k;
         unsigned int kb   = neighborZ[k];
         unsigned int ksw  = neighborY[kw];
         unsigned int kne  = k;
         unsigned int kse  = ks;
         unsigned int knw  = kw;
         unsigned int kbw  = neighborZ[kw];
         unsigned int kte  = k;
         unsigned int kbe  = kb;
         unsigned int ktw  = kw;
         unsigned int kbs  = neighborZ[ks];
         unsigned int ktn  = k;
         unsigned int kbn  = kb;
         unsigned int kts  = ks;
         unsigned int ktse = ks;
         unsigned int kbnw = kbw;
         unsigned int ktnw = kw;
         unsigned int kbse = kbs;
         unsigned int ktsw = ksw;
         unsigned int kbne = kb;
         unsigned int ktne = k;
         unsigned int kbsw = neighborZ[ksw];
         //////////////////////////////////////////////////////////////////////////
         real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

         (D.f[d000])[kzero] =   c8o27* (drho-cu_sq*(c1o1+drho));
         (D.f[dP00   ])[ke   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq));
         (D.f[dM00   ])[kw   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq));
         (D.f[d0P0   ])[kn   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq));
         (D.f[d0M0   ])[ks   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
         (D.f[d00P   ])[kt   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq));
         (D.f[d00M   ])[kb   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq));
         (D.f[dPP0  ])[kne  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
         (D.f[dMM0  ])[ksw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
         (D.f[dPM0  ])[kse  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
         (D.f[dMP0  ])[knw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
         (D.f[dP0P  ])[kte  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
         (D.f[dM0M  ])[kbw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
         (D.f[dP0M  ])[kbe  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
         (D.f[dM0P  ])[ktw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
         (D.f[d0PP  ])[ktn  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
         (D.f[d0MM  ])[kbs  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
         (D.f[d0PM  ])[kbn  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
         (D.f[d0MP  ])[kts  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
         (D.f[dPPP ])[ktne ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
         (D.f[dMMM ])[kbsw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
         (D.f[dPPM ])[kbne ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
         (D.f[dMMP ])[ktsw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
         (D.f[dPMP ])[ktse ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
         (D.f[dMPM ])[kbnw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
         (D.f[dPMM ])[kbse ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
         (D.f[dMPP ])[ktnw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));
      }
   }
}










////////////////////////////////////////////////////////////////////////////////
__global__ void InitNavierStokesCompressibleNonEquilibrium_Device( unsigned int* neighborX,
                                                   unsigned int* neighborY,
                                                   unsigned int* neighborZ,
                                                   unsigned int* neighborWSB,
                                                   unsigned int* geoD,
                                                   real* rho,
                                                   real* ux,
                                                   real* uy,
                                                   real* uz,
                                                   unsigned int size_Mat,
                                                   real* DD,
                                                   real omega,
                                                   bool EvenOrOdd)
{
    ////////////////////////////////////////////////////////////////////////////////
    const unsigned  x = threadIdx.x;  // Globaler x-Index 
    const unsigned  y = blockIdx.x;   // Globaler y-Index 
    const unsigned  z = blockIdx.y;   // Globaler z-Index 
    
    const unsigned nx = blockDim.x;
    const unsigned ny = gridDim.x;
    
    const unsigned k = nx*(ny*z + y) + x;
    //////////////////////////////////////////////////////////////////////////

    if(k<size_Mat)
    {
        ////////////////////////////////////////////////////////////////////////////////
        unsigned int BC;
        BC = geoD[k];

        if( BC != GEO_SOLID &&  BC != GEO_VOID)
        {
            Distributions27 D;
            if (EvenOrOdd==true)
            {
                D.f[dP00   ] = &DD[dP00   *size_Mat];
                D.f[dM00   ] = &DD[dM00   *size_Mat];
                D.f[d0P0   ] = &DD[d0P0   *size_Mat];
                D.f[d0M0   ] = &DD[d0M0   *size_Mat];
                D.f[d00P   ] = &DD[d00P   *size_Mat];
                D.f[d00M   ] = &DD[d00M   *size_Mat];
                D.f[dPP0  ] = &DD[dPP0  *size_Mat];
                D.f[dMM0  ] = &DD[dMM0  *size_Mat];
                D.f[dPM0  ] = &DD[dPM0  *size_Mat];
                D.f[dMP0  ] = &DD[dMP0  *size_Mat];
                D.f[dP0P  ] = &DD[dP0P  *size_Mat];
                D.f[dM0M  ] = &DD[dM0M  *size_Mat];
                D.f[dP0M  ] = &DD[dP0M  *size_Mat];
                D.f[dM0P  ] = &DD[dM0P  *size_Mat];
                D.f[d0PP  ] = &DD[d0PP  *size_Mat];
                D.f[d0MM  ] = &DD[d0MM  *size_Mat];
                D.f[d0PM  ] = &DD[d0PM  *size_Mat];
                D.f[d0MP  ] = &DD[d0MP  *size_Mat];
                D.f[d000] = &DD[d000*size_Mat];
                D.f[dPPP ] = &DD[dPPP *size_Mat];
                D.f[dMMP ] = &DD[dMMP *size_Mat];
                D.f[dPMP ] = &DD[dPMP *size_Mat];
                D.f[dMPP ] = &DD[dMPP *size_Mat];
                D.f[dPPM ] = &DD[dPPM *size_Mat];
                D.f[dMMM ] = &DD[dMMM *size_Mat];
                D.f[dPMM ] = &DD[dPMM *size_Mat];
                D.f[dMPM ] = &DD[dMPM *size_Mat];
            }
            else
            {
                D.f[dM00   ] = &DD[dP00   *size_Mat];
                D.f[dP00   ] = &DD[dM00   *size_Mat];
                D.f[d0M0   ] = &DD[d0P0   *size_Mat];
                D.f[d0P0   ] = &DD[d0M0   *size_Mat];
                D.f[d00M   ] = &DD[d00P   *size_Mat];
                D.f[d00P   ] = &DD[d00M   *size_Mat];
                D.f[dMM0  ] = &DD[dPP0  *size_Mat];
                D.f[dPP0  ] = &DD[dMM0  *size_Mat];
                D.f[dMP0  ] = &DD[dPM0  *size_Mat];
                D.f[dPM0  ] = &DD[dMP0  *size_Mat];
                D.f[dM0M  ] = &DD[dP0P  *size_Mat];
                D.f[dP0P  ] = &DD[dM0M  *size_Mat];
                D.f[dM0P  ] = &DD[dP0M  *size_Mat];
                D.f[dP0M  ] = &DD[dM0P  *size_Mat];
                D.f[d0MM  ] = &DD[d0PP  *size_Mat];
                D.f[d0PP  ] = &DD[d0MM  *size_Mat];
                D.f[d0MP  ] = &DD[d0PM  *size_Mat];
                D.f[d0PM  ] = &DD[d0MP  *size_Mat];
                D.f[d000] = &DD[d000*size_Mat];
                D.f[dMMM ] = &DD[dPPP *size_Mat];
                D.f[dPPM ] = &DD[dMMP *size_Mat];
                D.f[dMPM ] = &DD[dPMP *size_Mat];
                D.f[dPMM ] = &DD[dMPP *size_Mat];
                D.f[dMMP ] = &DD[dPPM *size_Mat];
                D.f[dPPP ] = &DD[dMMM *size_Mat];
                D.f[dMPP ] = &DD[dPMM *size_Mat];
                D.f[dPMP ] = &DD[dMPM *size_Mat];
            }
            //////////////////////////////////////////////////////////////////////////
            real drho = rho[k];//0.0f;//
            real  vx1 = ux[k]; //0.0f;//
            real  vx2 = uy[k]; //0.0f;//
            real  vx3 = uz[k]; //0.0f;//
            //////////////////////////////////////////////////////////////////////////
            //index
            //////////////////////////////////////////////////////////////////////////
            unsigned int kzero= k;
            unsigned int ke   = k;
            unsigned int kw   = neighborX[k];
            unsigned int kn   = k;
            unsigned int ks   = neighborY[k];
            unsigned int kt   = k;
            unsigned int kb   = neighborZ[k];
            unsigned int ksw  = neighborY[kw];
            unsigned int kne  = k;
            unsigned int kse  = ks;
            unsigned int knw  = kw;
            unsigned int kbw  = neighborZ[kw];
            unsigned int kte  = k;
            unsigned int kbe  = kb;
            unsigned int ktw  = kw;
            unsigned int kbs  = neighborZ[ks];
            unsigned int ktn  = k;
            unsigned int kbn  = kb;
            unsigned int kts  = ks;
            unsigned int ktse = ks;
            unsigned int kbnw = kbw;
            unsigned int ktnw = kw;
            unsigned int kbse = kbs;
            unsigned int ktsw = ksw;
            unsigned int kbne = kb;
            unsigned int ktne = k;
            unsigned int kbsw = neighborZ[ksw];
            //////////////////////////////////////////////////////////////////////////////
            //neighbor index
            uint kPx   = neighborX[k];
            uint kPy   = neighborY[k];
            uint kPz   = neighborZ[k];
            uint kMxyz = neighborWSB[k];
            uint kMx   = neighborZ[neighborY[kMxyz]];
            uint kMy   = neighborZ[neighborX[kMxyz]];
            uint kMz   = neighborY[neighborX[kMxyz]];
            //////////////////////////////////////////////////////////////////////////
            //getVeloX//
            real vx1NeighborPx = ux[kPx];
            real vx1NeighborMx = ux[kMx];
            real vx1NeighborPy = ux[kPy];
            real vx1NeighborMy = ux[kMy];
            real vx1NeighborPz = ux[kPz];
            real vx1NeighborMz = ux[kMz];
            //getVeloY//
            real vx2NeighborPx = uy[kPx];
            real vx2NeighborMx = uy[kMx];
            real vx2NeighborPy = uy[kPy];
            real vx2NeighborMy = uy[kMy];
            real vx2NeighborPz = uy[kPz];
            real vx2NeighborMz = uy[kMz];
            //getVeloZ//
            real vx3NeighborPx = uz[kPx];
            real vx3NeighborMx = uz[kMx];
            real vx3NeighborPy = uz[kPy];
            real vx3NeighborMy = uz[kMy];
            real vx3NeighborPz = uz[kPz];
            real vx3NeighborMz = uz[kMz];
            //////////////////////////////////////////////////////////////////////////

            real dvx1dx = (vx1NeighborPx - vx1NeighborMx) / c2o1;
            real dvx1dy = (vx1NeighborPy - vx1NeighborMy) / c2o1;
            real dvx1dz = (vx1NeighborPz - vx1NeighborMz) / c2o1;

            real dvx2dx = (vx2NeighborPx - vx2NeighborMx) / c2o1;
            real dvx2dy = (vx2NeighborPy - vx2NeighborMy) / c2o1;
            real dvx2dz = (vx2NeighborPz - vx2NeighborMz) / c2o1;

            real dvx3dx = (vx3NeighborPx - vx3NeighborMx) / c2o1;
            real dvx3dy = (vx3NeighborPy - vx3NeighborMy) / c2o1;
            real dvx3dz = (vx3NeighborPz - vx3NeighborMz) / c2o1;

            //////////////////////////////////////////////////////////////////////////

            // the following code is copy and pasted from VirtualFluidsVisitors/InitDistributionsBlockVisitor.cpp
            // i.e. Konstantins code

            real ax = dvx1dx;
            real ay = dvx1dy;
            real az = dvx1dz;

            real bx = dvx2dx;
            real by = dvx2dy;
            real bz = dvx2dz;

            real cx = dvx3dx;
            real cy = dvx3dy;
            real cz = dvx3dz;

            real eps_new = c1o1;
            real op      = c1o1;
            real o       = omega;

            real f_E    =            eps_new *((5.*ax*o + 5.*by*o + 5.*cz*o - 8.*ax*op + 4.*by*op + 4.*cz*op)/(54.*o*op));

            real f_N    =    f_E   + eps_new *((2.*(ax - by))/(9.*o));
            real f_T    =    f_E   + eps_new *((2.*(ax - cz))/(9.*o));
            real f_NE   =            eps_new *(-(5.*cz*o + 3.*(ay + bx)*op - 2.*cz*op + ax*(5.*o + op) + by*(5.*o + op))/(54.*o*op));
            real f_SE   =    f_NE  + eps_new *((  ay + bx )/(9.*o));
            real f_TE   =            eps_new *(-(5.*cz*o + by*(5.*o - 2.*op) + 3.*(az + cx)*op + cz*op + ax*(5.*o + op))/(54.*o*op));
            real f_BE   =    f_TE  + eps_new *((  az + cx )/(9.*o));
            real f_TN   =            eps_new *(-(5.*ax*o + 5.*by*o + 5.*cz*o - 2.*ax*op + by*op + 3.*bz*op + 3.*cy*op + cz*op)/(54.*o*op));
            real f_BN   =    f_TN  + eps_new *((  bz + cy )/(9.*o));
            real f_ZERO =            eps_new *((5.*(ax + by + cz))/(9.*op));
            real f_TNE  =            eps_new *(-(ay + az + bx + bz + cx + cy)/(72.*o));
            real f_TSW  =  - f_TNE - eps_new *((ay + bx)/(36.*o));
            real f_TSE  =  - f_TNE - eps_new *((az + cx)/(36.*o));
            real f_TNW  =  - f_TNE - eps_new *((bz + cy)/(36.*o));

            //////////////////////////////////////////////////////////////////////////
            real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

            (D.f[d000])[kzero] =   c8o27* (drho-cu_sq*(c1o1+drho));
            (D.f[dP00   ])[ke   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq));
            (D.f[dM00   ])[kw   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq));
            (D.f[d0P0   ])[kn   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq));
            (D.f[d0M0   ])[ks   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
            (D.f[d00P   ])[kt   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq));
            (D.f[d00M   ])[kb   ] =   c2o27* (drho+ (c1o1+drho) * (c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq));
            (D.f[dPP0  ])[kne  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
            (D.f[dMM0  ])[ksw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
            (D.f[dPM0  ])[kse  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
            (D.f[dMP0  ])[knw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
            (D.f[dP0P  ])[kte  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
            (D.f[dM0M  ])[kbw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
            (D.f[dP0M  ])[kbe  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
            (D.f[dM0P  ])[ktw  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
            (D.f[d0PP  ])[ktn  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
            (D.f[d0MM  ])[kbs  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
            (D.f[d0PM  ])[kbn  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
            (D.f[d0MP  ])[kts  ] =   c1o54* (drho+ (c1o1+drho) * (c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
            (D.f[dPPP ])[ktne ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
            (D.f[dMMM ])[kbsw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
            (D.f[dPPM ])[kbne ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
            (D.f[dMMP ])[ktsw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
            (D.f[dPMP ])[ktse ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
            (D.f[dMPM ])[kbnw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
            (D.f[dPMM ])[kbse ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
            (D.f[dMPP ])[ktnw ] =   c1o216*(drho+ (c1o1+drho) * (c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));

            //////////////////////////////////////////////////////////////////////////

            (D.f[d000])[kzero] += (c1o1+drho) * f_ZERO;
            (D.f[dP00   ])[ke   ] += (c1o1+drho) * f_E   ;
            (D.f[dM00   ])[kw   ] += (c1o1+drho) * f_E   ;
            (D.f[d0P0   ])[kn   ] += (c1o1+drho) * f_N   ;
            (D.f[d0M0   ])[ks   ] += (c1o1+drho) * f_N   ;
            (D.f[d00P   ])[kt   ] += (c1o1+drho) * f_T   ;
            (D.f[d00M   ])[kb   ] += (c1o1+drho) * f_T   ;
            (D.f[dPP0  ])[kne  ] += (c1o1+drho) * f_NE  ;
            (D.f[dMM0  ])[ksw  ] += (c1o1+drho) * f_NE  ;
            (D.f[dPM0  ])[kse  ] += (c1o1+drho) * f_SE  ;
            (D.f[dMP0  ])[knw  ] += (c1o1+drho) * f_SE  ;
            (D.f[dP0P  ])[kte  ] += (c1o1+drho) * f_TE  ;
            (D.f[dM0M  ])[kbw  ] += (c1o1+drho) * f_TE  ;
            (D.f[dP0M  ])[kbe  ] += (c1o1+drho) * f_BE  ;
            (D.f[dM0P  ])[ktw  ] += (c1o1+drho) * f_BE  ;
            (D.f[d0PP  ])[ktn  ] += (c1o1+drho) * f_TN  ;
            (D.f[d0MM  ])[kbs  ] += (c1o1+drho) * f_TN  ;
            (D.f[d0PM  ])[kbn  ] += (c1o1+drho) * f_BN  ;
            (D.f[d0MP  ])[kts  ] += (c1o1+drho) * f_BN  ;
            (D.f[dPPP ])[ktne ] += (c1o1+drho) * f_TNE ;
            (D.f[dMMM ])[kbsw ] += (c1o1+drho) * f_TNE ;
            (D.f[dPPM ])[kbne ] += (c1o1+drho) * f_TSW ;
            (D.f[dMMP ])[ktsw ] += (c1o1+drho) * f_TSW ;
            (D.f[dPMP ])[ktse ] += (c1o1+drho) * f_TSE ;
            (D.f[dMPM ])[kbnw ] += (c1o1+drho) * f_TSE ;
            (D.f[dPMM ])[kbse ] += (c1o1+drho) * f_TNW ;
            (D.f[dMPP ])[ktnw ] += (c1o1+drho) * f_TNW ;

            //////////////////////////////////////////////////////////////////////////
        }
        else
        {
            //////////////////////////////////////////////////////////////////////////
            Distributions27 D;
            D.f[d000] = &DD[d000*size_Mat];
            //////////////////////////////////////////////////////////////////////////
            (D.f[d000])[k] = c96o1;
            //////////////////////////////////////////////////////////////////////////
        }
   }
}