#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \author Martin Schoenherr
//=======================================================================================
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;
#include "math.h"

__global__ void InitAdvectionDiffusionIncompressible_Device(
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned int* geoD,
    real* Conc,
    real* ux,
    real* uy,
    real* uz,
    unsigned int size_Mat,
    real* DD27,
    bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   geoD[k];

      if( BC != GEO_SOLID && BC != GEO_VOID)
      {
         Distributions27 D27;
         if (EvenOrOdd==true)
         {
            D27.f[dP00   ] = &DD27[dP00   *size_Mat];
            D27.f[dM00   ] = &DD27[dM00   *size_Mat];
            D27.f[d0P0   ] = &DD27[d0P0   *size_Mat];
            D27.f[d0M0   ] = &DD27[d0M0   *size_Mat];
            D27.f[d00P   ] = &DD27[d00P   *size_Mat];
            D27.f[d00M   ] = &DD27[d00M   *size_Mat];
            D27.f[dPP0  ] = &DD27[dPP0  *size_Mat];
            D27.f[dMM0  ] = &DD27[dMM0  *size_Mat];
            D27.f[dPM0  ] = &DD27[dPM0  *size_Mat];
            D27.f[dMP0  ] = &DD27[dMP0  *size_Mat];
            D27.f[dP0P  ] = &DD27[dP0P  *size_Mat];
            D27.f[dM0M  ] = &DD27[dM0M  *size_Mat];
            D27.f[dP0M  ] = &DD27[dP0M  *size_Mat];
            D27.f[dM0P  ] = &DD27[dM0P  *size_Mat];
            D27.f[d0PP  ] = &DD27[d0PP  *size_Mat];
            D27.f[d0MM  ] = &DD27[d0MM  *size_Mat];
            D27.f[d0PM  ] = &DD27[d0PM  *size_Mat];
            D27.f[d0MP  ] = &DD27[d0MP  *size_Mat];
            D27.f[d000] = &DD27[d000*size_Mat];
            D27.f[dPPP ] = &DD27[dPPP *size_Mat];
            D27.f[dMMP ] = &DD27[dMMP *size_Mat];
            D27.f[dPMP ] = &DD27[dPMP *size_Mat];
            D27.f[dMPP ] = &DD27[dMPP *size_Mat];
            D27.f[dPPM ] = &DD27[dPPM *size_Mat];
            D27.f[dMMM ] = &DD27[dMMM *size_Mat];
            D27.f[dPMM ] = &DD27[dPMM *size_Mat];
            D27.f[dMPM ] = &DD27[dMPM *size_Mat];
         }
         else
         {
            D27.f[dM00   ] = &DD27[dP00   *size_Mat];
            D27.f[dP00   ] = &DD27[dM00   *size_Mat];
            D27.f[d0M0   ] = &DD27[d0P0   *size_Mat];
            D27.f[d0P0   ] = &DD27[d0M0   *size_Mat];
            D27.f[d00M   ] = &DD27[d00P   *size_Mat];
            D27.f[d00P   ] = &DD27[d00M   *size_Mat];
            D27.f[dMM0  ] = &DD27[dPP0  *size_Mat];
            D27.f[dPP0  ] = &DD27[dMM0  *size_Mat];
            D27.f[dMP0  ] = &DD27[dPM0  *size_Mat];
            D27.f[dPM0  ] = &DD27[dMP0  *size_Mat];
            D27.f[dM0M  ] = &DD27[dP0P  *size_Mat];
            D27.f[dP0P  ] = &DD27[dM0M  *size_Mat];
            D27.f[dM0P  ] = &DD27[dP0M  *size_Mat];
            D27.f[dP0M  ] = &DD27[dM0P  *size_Mat];
            D27.f[d0MM  ] = &DD27[d0PP  *size_Mat];
            D27.f[d0PP  ] = &DD27[d0MM  *size_Mat];
            D27.f[d0MP  ] = &DD27[d0PM  *size_Mat];
            D27.f[d0PM  ] = &DD27[d0MP  *size_Mat];
            D27.f[d000] = &DD27[d000*size_Mat];
            D27.f[dMMM ] = &DD27[dPPP *size_Mat];
            D27.f[dPPM ] = &DD27[dMMP *size_Mat];
            D27.f[dMPM ] = &DD27[dPMP *size_Mat];
            D27.f[dPMM ] = &DD27[dMPP *size_Mat];
            D27.f[dMMP ] = &DD27[dPPM *size_Mat];
            D27.f[dPPP ] = &DD27[dMMM *size_Mat];
            D27.f[dMPP ] = &DD27[dPMM *size_Mat];
            D27.f[dPMP ] = &DD27[dMPM *size_Mat];
         }
         //////////////////////////////////////////////////////////////////////////
         real ConcD = Conc[k];
         real   vx1 = ux[k];
         real   vx2 = uy[k];
         real   vx3 = uz[k];
         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         //D3Q27
         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         //index
         unsigned int kzero= k;
         unsigned int ke   = k;
         unsigned int kw   = neighborX[k];
         unsigned int kn   = k;
         unsigned int ks   = neighborY[k];
         unsigned int kt   = k;
         unsigned int kb   = neighborZ[k];
         unsigned int ksw  = neighborY[kw];
         unsigned int kne  = k;
         unsigned int kse  = ks;
         unsigned int knw  = kw;
         unsigned int kbw  = neighborZ[kw];
         unsigned int kte  = k;
         unsigned int kbe  = kb;
         unsigned int ktw  = kw;
         unsigned int kbs  = neighborZ[ks];
         unsigned int ktn  = k;
         unsigned int kbn  = kb;
         unsigned int kts  = ks;
         unsigned int ktse = ks;
         unsigned int kbnw = kbw;
         unsigned int ktnw = kw;
         unsigned int kbse = kbs;
         unsigned int ktsw = ksw;
         unsigned int kbne = kb;
         unsigned int ktne = k;
         unsigned int kbsw = neighborZ[ksw];
         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

         (D27.f[d000])[kzero] =   c8o27* ConcD*(c1o1-cu_sq);
         (D27.f[dP00   ])[ke   ] =   c2o27* ConcD*(c1o1+c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq);
         (D27.f[dM00   ])[kw   ] =   c2o27* ConcD*(c1o1+c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq);
         (D27.f[d0P0   ])[kn   ] =   c2o27* ConcD*(c1o1+c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq);
         (D27.f[d0M0   ])[ks   ] =   c2o27* ConcD*(c1o1+c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq);
         (D27.f[d00P   ])[kt   ] =   c2o27* ConcD*(c1o1+c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq);
         (D27.f[d00M   ])[kb   ] =   c2o27* ConcD*(c1o1+c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq);
         (D27.f[dPP0  ])[kne  ] =   c1o54* ConcD*(c1o1+c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq);
         (D27.f[dMM0  ])[ksw  ] =   c1o54* ConcD*(c1o1+c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq);
         (D27.f[dPM0  ])[kse  ] =   c1o54* ConcD*(c1o1+c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq);
         (D27.f[dMP0  ])[knw  ] =   c1o54* ConcD*(c1o1+c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq);
         (D27.f[dP0P  ])[kte  ] =   c1o54* ConcD*(c1o1+c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq);
         (D27.f[dM0M  ])[kbw  ] =   c1o54* ConcD*(c1o1+c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq);
         (D27.f[dP0M  ])[kbe  ] =   c1o54* ConcD*(c1o1+c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq);
         (D27.f[dM0P  ])[ktw  ] =   c1o54* ConcD*(c1o1+c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq);
         (D27.f[d0PP  ])[ktn  ] =   c1o54* ConcD*(c1o1+c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq);
         (D27.f[d0MM  ])[kbs  ] =   c1o54* ConcD*(c1o1+c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq);
         (D27.f[d0PM  ])[kbn  ] =   c1o54* ConcD*(c1o1+c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq);
         (D27.f[d0MP  ])[kts  ] =   c1o54* ConcD*(c1o1+c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq);
         (D27.f[dPPP ])[ktne ] =   c1o216*ConcD*(c1o1+c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq);
         (D27.f[dMMM ])[kbsw ] =   c1o216*ConcD*(c1o1+c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq);
         (D27.f[dPPM ])[kbne ] =   c1o216*ConcD*(c1o1+c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq);
         (D27.f[dMMP ])[ktsw ] =   c1o216*ConcD*(c1o1+c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq);
         (D27.f[dPMP ])[ktse ] =   c1o216*ConcD*(c1o1+c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq);
         (D27.f[dMPM ])[kbnw ] =   c1o216*ConcD*(c1o1+c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq);
         (D27.f[dPMM ])[kbse ] =   c1o216*ConcD*(c1o1+c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq);
         (D27.f[dMPP ])[ktnw ] =   c1o216*ConcD*(c1o1+c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq);
         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      }
   }
}