#include "hip/hip_runtime.h"
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;


__global__ void LB_Init_Comp_AD_27(unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned int* geoD,
    real* Conc,
    real* ux,
    real* uy,
    real* uz,
    unsigned int size_Mat,
    real* DD27,
    bool EvenOrOdd)
{
    ////////////////////////////////////////////////////////////////////////////////
    const unsigned  x = threadIdx.x;  // Globaler x-Index 
    const unsigned  y = blockIdx.x;   // Globaler y-Index 
    const unsigned  z = blockIdx.y;   // Globaler z-Index 

    const unsigned nx = blockDim.x;
    const unsigned ny = gridDim.x;

    const unsigned k = nx*(ny*z + y) + x;
    //////////////////////////////////////////////////////////////////////////

    if (k<size_Mat)
    {
        ////////////////////////////////////////////////////////////////////////////////
        unsigned int BC;
        BC = geoD[k];

        if (BC != GEO_SOLID && BC != GEO_VOID)
        {
            Distributions27 D27;
            if (EvenOrOdd == true)
            {
                D27.f[dP00] = &DD27[dP00   *size_Mat];
                D27.f[dM00] = &DD27[dM00   *size_Mat];
                D27.f[d0P0] = &DD27[d0P0   *size_Mat];
                D27.f[d0M0] = &DD27[d0M0   *size_Mat];
                D27.f[d00P] = &DD27[d00P   *size_Mat];
                D27.f[d00M] = &DD27[d00M   *size_Mat];
                D27.f[dPP0] = &DD27[dPP0  *size_Mat];
                D27.f[dMM0] = &DD27[dMM0  *size_Mat];
                D27.f[dPM0] = &DD27[dPM0  *size_Mat];
                D27.f[dMP0] = &DD27[dMP0  *size_Mat];
                D27.f[dP0P] = &DD27[dP0P  *size_Mat];
                D27.f[dM0M] = &DD27[dM0M  *size_Mat];
                D27.f[dP0M] = &DD27[dP0M  *size_Mat];
                D27.f[dM0P] = &DD27[dM0P  *size_Mat];
                D27.f[d0PP] = &DD27[d0PP  *size_Mat];
                D27.f[d0MM] = &DD27[d0MM  *size_Mat];
                D27.f[d0PM] = &DD27[d0PM  *size_Mat];
                D27.f[d0MP] = &DD27[d0MP  *size_Mat];
                D27.f[d000] = &DD27[d000*size_Mat];
                D27.f[dPPP] = &DD27[dPPP *size_Mat];
                D27.f[dMMP] = &DD27[dMMP *size_Mat];
                D27.f[dPMP] = &DD27[dPMP *size_Mat];
                D27.f[dMPP] = &DD27[dMPP *size_Mat];
                D27.f[dPPM] = &DD27[dPPM *size_Mat];
                D27.f[dMMM] = &DD27[dMMM *size_Mat];
                D27.f[dPMM]= &DD27[dPMM *size_Mat];
                D27.f[dMPM]= &DD27[dMPM *size_Mat];
            }
            else
            {
                D27.f[dM00] = &DD27[dP00   *size_Mat];
                D27.f[dP00] = &DD27[dM00   *size_Mat];
                D27.f[d0M0] = &DD27[d0P0   *size_Mat];
                D27.f[d0P0] = &DD27[d0M0   *size_Mat];
                D27.f[d00M] = &DD27[d00P   *size_Mat];
                D27.f[d00P] = &DD27[d00M   *size_Mat];
                D27.f[dMM0] = &DD27[dPP0  *size_Mat];
                D27.f[dPP0] = &DD27[dMM0  *size_Mat];
                D27.f[dMP0] = &DD27[dPM0  *size_Mat];
                D27.f[dPM0] = &DD27[dMP0  *size_Mat];
                D27.f[dM0M] = &DD27[dP0P  *size_Mat];
                D27.f[dP0P] = &DD27[dM0M  *size_Mat];
                D27.f[dM0P] = &DD27[dP0M  *size_Mat];
                D27.f[dP0M] = &DD27[dM0P  *size_Mat];
                D27.f[d0MM] = &DD27[d0PP  *size_Mat];
                D27.f[d0PP] = &DD27[d0MM  *size_Mat];
                D27.f[d0MP] = &DD27[d0PM  *size_Mat];
                D27.f[d0PM] = &DD27[d0MP  *size_Mat];
                D27.f[d000] = &DD27[d000*size_Mat];
                D27.f[dMMM] = &DD27[dPPP *size_Mat];
                D27.f[dPPM] = &DD27[dMMP *size_Mat];
                D27.f[dMPM]= &DD27[dPMP *size_Mat];
                D27.f[dPMM]= &DD27[dMPP *size_Mat];
                D27.f[dMMP] = &DD27[dPPM *size_Mat];
                D27.f[dPPP] = &DD27[dMMM *size_Mat];
                D27.f[dMPP] = &DD27[dPMM *size_Mat];
                D27.f[dPMP] = &DD27[dMPM *size_Mat];
            }
            //////////////////////////////////////////////////////////////////////////
            real ConcD = Conc[k];
            real   vx1 = ux[k];
            real   vx2 = uy[k];
            real   vx3 = uz[k];
            //real lambdaD     = -three + sqrt(three);
            //real Diffusivity = c1o20;
            //real Lam         = -(c1o2+one/lambdaD);
            //real nue_d       = Lam/three;
            //real ae          = Diffusivity/nue_d - one;
            //real ux_sq       = vx1 * vx1;
            //real uy_sq       = vx2 * vx2;
            //real uz_sq       = vx3 * vx3;
            ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //D3Q7
            ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //index
            //unsigned int kzero= k;
            //unsigned int ke   = k;
            //unsigned int kw   = neighborX[k];
            //unsigned int kn   = k;
            //unsigned int ks   = neighborY[k];
            //unsigned int kt   = k;
            //unsigned int kb   = neighborZ[k];
            //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //(D7.f[0])[kzero] = ConcD*(c1o3*(ae*(-three))-(ux_sq+uy_sq+uz_sq));
            //(D7.f[1])[ke   ] = ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)+vx1*c1o2);
            //(D7.f[2])[kw   ] = ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)-vx1*c1o2);
            //(D7.f[3])[kn   ] = ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)+vx2*c1o2);
            //(D7.f[4])[ks   ] = ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)-vx2*c1o2);
            //(D7.f[5])[kt   ] = ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)+vx3*c1o2);
            //(D7.f[6])[kb   ] = ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)-vx3*c1o2);
            ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


            ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //D3Q27
            ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //index
            unsigned int kzero = k;
            unsigned int ke = k;
            unsigned int kw = neighborX[k];
            unsigned int kn = k;
            unsigned int ks = neighborY[k];
            unsigned int kt = k;
            unsigned int kb = neighborZ[k];
            unsigned int ksw = neighborY[kw];
            unsigned int kne = k;
            unsigned int kse = ks;
            unsigned int knw = kw;
            unsigned int kbw = neighborZ[kw];
            unsigned int kte = k;
            unsigned int kbe = kb;
            unsigned int ktw = kw;
            unsigned int kbs = neighborZ[ks];
            unsigned int ktn = k;
            unsigned int kbn = kb;
            unsigned int kts = ks;
            unsigned int ktse = ks;
            unsigned int kbnw = kbw;
            unsigned int ktnw = kw;
            unsigned int kbse = kbs;
            unsigned int ktsw = ksw;
            unsigned int kbne = kb;
            unsigned int ktne = k;
            unsigned int kbsw = neighborZ[ksw];
            ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            real cu_sq = c3o2*(vx1*vx1 + vx2*vx2 + vx3*vx3);

            (D27.f[d000])[kzero] = c8o27* ConcD*(c1o1 - cu_sq);
            (D27.f[dP00])[ke] = c2o27* ConcD*(c1o1 + c3o1*(vx1)+c9o2*(vx1)*(vx1)-cu_sq);
            (D27.f[dM00])[kw] = c2o27* ConcD*(c1o1 + c3o1*(-vx1) + c9o2*(-vx1)*(-vx1) - cu_sq);
            (D27.f[d0P0])[kn] = c2o27* ConcD*(c1o1 + c3o1*(vx2)+c9o2*(vx2)*(vx2)-cu_sq);
            (D27.f[d0M0])[ks] = c2o27* ConcD*(c1o1 + c3o1*(-vx2) + c9o2*(-vx2)*(-vx2) - cu_sq);
            (D27.f[d00P])[kt] = c2o27* ConcD*(c1o1 + c3o1*(vx3)+c9o2*(vx3)*(vx3)-cu_sq);
            (D27.f[d00M])[kb] = c2o27* ConcD*(c1o1 + c3o1*(-vx3) + c9o2*(-vx3)*(-vx3) - cu_sq);
            (D27.f[dPP0])[kne] = c1o54* ConcD*(c1o1 + c3o1*(vx1 + vx2) + c9o2*(vx1 + vx2)*(vx1 + vx2) - cu_sq);
            (D27.f[dMM0])[ksw] = c1o54* ConcD*(c1o1 + c3o1*(-vx1 - vx2) + c9o2*(-vx1 - vx2)*(-vx1 - vx2) - cu_sq);
            (D27.f[dPM0])[kse] = c1o54* ConcD*(c1o1 + c3o1*(vx1 - vx2) + c9o2*(vx1 - vx2)*(vx1 - vx2) - cu_sq);
            (D27.f[dMP0])[knw] = c1o54* ConcD*(c1o1 + c3o1*(-vx1 + vx2) + c9o2*(-vx1 + vx2)*(-vx1 + vx2) - cu_sq);
            (D27.f[dP0P])[kte] = c1o54* ConcD*(c1o1 + c3o1*(vx1 + vx3) + c9o2*(vx1 + vx3)*(vx1 + vx3) - cu_sq);
            (D27.f[dM0M])[kbw] = c1o54* ConcD*(c1o1 + c3o1*(-vx1 - vx3) + c9o2*(-vx1 - vx3)*(-vx1 - vx3) - cu_sq);
            (D27.f[dP0M])[kbe] = c1o54* ConcD*(c1o1 + c3o1*(vx1 - vx3) + c9o2*(vx1 - vx3)*(vx1 - vx3) - cu_sq);
            (D27.f[dM0P])[ktw] = c1o54* ConcD*(c1o1 + c3o1*(-vx1 + vx3) + c9o2*(-vx1 + vx3)*(-vx1 + vx3) - cu_sq);
            (D27.f[d0PP])[ktn] = c1o54* ConcD*(c1o1 + c3o1*(vx2 + vx3) + c9o2*(vx2 + vx3)*(vx2 + vx3) - cu_sq);
            (D27.f[d0MM])[kbs] = c1o54* ConcD*(c1o1 + c3o1*(-vx2 - vx3) + c9o2*(-vx2 - vx3)*(-vx2 - vx3) - cu_sq);
            (D27.f[d0PM])[kbn] = c1o54* ConcD*(c1o1 + c3o1*(vx2 - vx3) + c9o2*(vx2 - vx3)*(vx2 - vx3) - cu_sq);
            (D27.f[d0MP])[kts] = c1o54* ConcD*(c1o1 + c3o1*(-vx2 + vx3) + c9o2*(-vx2 + vx3)*(-vx2 + vx3) - cu_sq);
            (D27.f[dPPP])[ktne] = c1o216*ConcD*(c1o1 + c3o1*(vx1 + vx2 + vx3) + c9o2*(vx1 + vx2 + vx3)*(vx1 + vx2 + vx3) - cu_sq);
            (D27.f[dMMM])[kbsw] = c1o216*ConcD*(c1o1 + c3o1*(-vx1 - vx2 - vx3) + c9o2*(-vx1 - vx2 - vx3)*(-vx1 - vx2 - vx3) - cu_sq);
            (D27.f[dPPM])[kbne] = c1o216*ConcD*(c1o1 + c3o1*(vx1 + vx2 - vx3) + c9o2*(vx1 + vx2 - vx3)*(vx1 + vx2 - vx3) - cu_sq);
            (D27.f[dMMP])[ktsw] = c1o216*ConcD*(c1o1 + c3o1*(-vx1 - vx2 + vx3) + c9o2*(-vx1 - vx2 + vx3)*(-vx1 - vx2 + vx3) - cu_sq);
            (D27.f[dPMP])[ktse] = c1o216*ConcD*(c1o1 + c3o1*(vx1 - vx2 + vx3) + c9o2*(vx1 - vx2 + vx3)*(vx1 - vx2 + vx3) - cu_sq);
            (D27.f[dMPM])[kbnw] = c1o216*ConcD*(c1o1 + c3o1*(-vx1 + vx2 - vx3) + c9o2*(-vx1 + vx2 - vx3)*(-vx1 + vx2 - vx3) - cu_sq);
            (D27.f[dPMM])[kbse] = c1o216*ConcD*(c1o1 + c3o1*(vx1 - vx2 - vx3) + c9o2*(vx1 - vx2 - vx3)*(vx1 - vx2 - vx3) - cu_sq);
            (D27.f[dMPP])[ktnw] = c1o216*ConcD*(c1o1 + c3o1*(-vx1 + vx2 + vx3) + c9o2*(-vx1 + vx2 + vx3)*(-vx1 + vx2 + vx3) - cu_sq);
            ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        }
    }
}
