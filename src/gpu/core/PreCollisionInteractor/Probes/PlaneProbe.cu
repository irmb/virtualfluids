#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
//  for more details.
//
//  SPDX-License-Identifier: GPL-3.0-or-later
//  SPDX-FileCopyrightText: Copyright © VirtualFluids Project contributors, see AUTHORS.md in root folder
//
//! \addtogroup gpu_PreCollisionInteractor PreCollisionInteractor
//! \ingroup gpu_core core
//! \{
#include "Probe.h"
#include "PlaneProbe.h"

#include <cuda_helper/CudaGrid.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Parameter/Parameter.h"
#include "DataStructureInitializer/GridProvider.h"
#include "Cuda/CudaMemoryManager.h"


bool PlaneProbe::isAvailableStatistic(Statistic _variable)
{
    bool isAvailable;
    switch (_variable)
    {
        case Statistic::Instantaneous:
        case Statistic::Means:
        case Statistic::Variances:
            isAvailable = true;
            break;
        case Statistic::SpatialMeans:
        case Statistic::SpatioTemporalMeans:
        case Statistic::SpatialCovariances:
        case Statistic::SpatioTemporalCovariances:
        case Statistic::SpatialSkewness:
        case Statistic::SpatioTemporalSkewness:
        case Statistic::SpatialFlatness:
        case Statistic::SpatioTemporalFlatness:
            isAvailable = false;
            break;
        default:
            isAvailable = false;
    }
    return isAvailable;
}


std::vector<PostProcessingVariable> PlaneProbe::getPostProcessingVariables(Statistic statistic)
{
    std::vector<PostProcessingVariable> postProcessingVariables;
    switch (statistic)
    {
    case Statistic::Instantaneous:
        postProcessingVariables.push_back( PostProcessingVariable("vx",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho", this->densityRatio ) );
        break;
    case Statistic::Means:
        postProcessingVariables.push_back( PostProcessingVariable("vx_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho_mean", this->densityRatio ) );
        break;
    case Statistic::Variances:
        postProcessingVariables.push_back( PostProcessingVariable("vx_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho_var", this->densityRatio) );
        break;

    default:
        throw std::runtime_error("PlaneProbe::getPostProcessingVariables: Statistic unavailable!");
        break;
    }
    return postProcessingVariables;
}

void PlaneProbe::findPoints(std::vector<int>& probeIndices_level,
                            std::vector<real>& distX_level, std::vector<real>& distY_level, std::vector<real>& distZ_level,      
                            std::vector<real>& pointCoordsX_level, std::vector<real>& pointCoordsY_level, std::vector<real>& pointCoordsZ_level,
                            int level)
{
    real dx = abs(para->getParH(level)->coordinateX[1]-para->getParH(level)->coordinateX[para->getParH(level)->neighborX[1]]);
    for(size_t pos = 1; pos < para->getParH(level)->numberOfNodes; pos++ )
    {
        real pointCoordX = para->getParH(level)->coordinateX[pos];
        real pointCoordY = para->getParH(level)->coordinateY[pos];
        real pointCoordZ = para->getParH(level)->coordinateZ[pos];
        real distX = pointCoordX - this->posX;
        real distY = pointCoordY - this->posY;
        real distZ = pointCoordZ - this->posZ;

        if( distX <= this->deltaX && distY <= this->deltaY && distZ <= this->deltaZ &&
            distX >=0.f && distY >=0.f && distZ >=0.f)
        {
            probeIndices_level.push_back((int)pos);
            distX_level.push_back( distX/dx );
            distY_level.push_back( distY/dx );
            distZ_level.push_back( distZ/dx );
            pointCoordsX_level.push_back( pointCoordX );
            pointCoordsY_level.push_back( pointCoordY );
            pointCoordsZ_level.push_back( pointCoordZ );
        }
    }
}

void PlaneProbe::calculateQuantities(SPtr<ProbeStruct> probeStruct, uint t, int level)
{
    vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(para->getParH(level)->numberofthreads, probeStruct->nPoints);
    calcQuantitiesKernel<<<grid.grid, grid.threads>>>(  probeStruct->pointIndicesD,
                                                        probeStruct->nPoints,
                                                        0,
                                                        0,
                                                        probeStruct->timestepInTimeAverage,
                                                        probeStruct->nTimesteps,
                                                        para->getParD(level)->velocityX,
                                                        para->getParD(level)->velocityY,
                                                        para->getParD(level)->velocityZ,
                                                        para->getParD(level)->rho,
                                                        para->getParD(level)->neighborX,
                                                        para->getParD(level)->neighborY,
                                                        para->getParD(level)->neighborZ,
                                                        probeStruct->quantitiesD,
                                                        probeStruct->arrayOffsetsD,
                                                        probeStruct->quantitiesArrayD
                                                        );
}

void PlaneProbe::getTaggedFluidNodes(GridProvider* gridProvider)
{
    for(int level=0; level<=para->getMaxLevel(); level++)
    {
        SPtr<ProbeStruct> probeStruct = this->getProbeStruct(level);
        std::vector<uint> probeIndices( probeStruct->pointIndicesH, probeStruct->pointIndicesH+probeStruct->nIndices);
        gridProvider->tagFluidNodeIndices( probeIndices, CollisionTemplate::WriteMacroVars, level);
    }
}
//! \}
