#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
//  for more details.
//
//  SPDX-License-Identifier: GPL-3.0-or-later
//  SPDX-FileCopyrightText: Copyright © VirtualFluids Project contributors, see AUTHORS.md in root folder
//
//! \addtogroup gpu_PreCollisionInteractor PreCollisionInteractor
//! \ingroup gpu_core core
//! \{
#include "Probe.h"
#include "PointProbe.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cuda_helper/CudaGrid.h>

#include "Parameter/Parameter.h"
#include "DataStructureInitializer/GridProvider.h"
#include "Cuda/CudaMemoryManager.h"

bool PointProbe::isAvailableStatistic(Statistic _variable)
{
    bool isAvailable;
    switch (_variable)
    {
        case Statistic::Instantaneous:
        case Statistic::Means:
        case Statistic::Variances:
            isAvailable = true;
            break;
        case Statistic::SpatialMeans:
        case Statistic::SpatioTemporalMeans:
        case Statistic::SpatialCovariances:
        case Statistic::SpatioTemporalCovariances:
        case Statistic::SpatialSkewness:
        case Statistic::SpatioTemporalSkewness:
        case Statistic::SpatialFlatness:
        case Statistic::SpatioTemporalFlatness:
            isAvailable = false;
            break;
        default:
            isAvailable = false;
    }
    return isAvailable;
}

std::vector<PostProcessingVariable> PointProbe::getPostProcessingVariables(Statistic statistic)
{
    std::vector<PostProcessingVariable> postProcessingVariables;
    switch (statistic)
    {
    case Statistic::Instantaneous:
        postProcessingVariables.push_back( PostProcessingVariable("vx",  velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho", this->densityRatio ) );
        break;
    case Statistic::Means:
        postProcessingVariables.push_back( PostProcessingVariable("vx_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz_mean",  this->velocityRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho_mean", this->densityRatio ) );
        break;
    case Statistic::Variances:
        postProcessingVariables.push_back( PostProcessingVariable("vx_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vy_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("vz_var",  this->stressRatio) );
        postProcessingVariables.push_back( PostProcessingVariable("rho_var", this->densityRatio) );
        break;

    default:
        throw std::runtime_error("PointProbe::getPostProcessingVariables: Statistic unavailable!");
        break;
    }
    return postProcessingVariables;
}

void PointProbe::findPoints(std::vector<int>& probeIndices_level,
                       std::vector<real>& distX_level, std::vector<real>& distY_level, std::vector<real>& distZ_level,      
                       std::vector<real>& pointCoordsX_level, std::vector<real>& pointCoordsY_level, std::vector<real>& pointCoordsZ_level,
                       int level)
{

    real dx = abs(para->getParH(level)->coordinateX[1]-para->getParH(level)->coordinateX[para->getParH(level)->neighborX[1]]);
    for(size_t pos = 1; pos < para->getParH(level)->numberOfNodes; pos++ )
    {    
        for(uint point=0; point<this->pointCoordsX.size(); point++)
        {
            real pointCoordX = this->pointCoordsX[point];
            real pointCoordY = this->pointCoordsY[point];
            real pointCoordZ = this->pointCoordsZ[point];
            real distX = pointCoordX-para->getParH(level)->coordinateX[pos];
            real distY = pointCoordY-para->getParH(level)->coordinateY[pos];
            real distZ = pointCoordZ-para->getParH(level)->coordinateZ[pos];
            if( distX <=dx && distY <=dx && distZ <=dx &&
                distX >0.f && distY >0.f && distZ >0.f)
            {
                probeIndices_level.push_back((int)pos);
                distX_level.push_back( distX/dx );
                distY_level.push_back( distY/dx );
                distZ_level.push_back( distZ/dx );
                pointCoordsX_level.push_back( pointCoordX );
                pointCoordsY_level.push_back( pointCoordY );
                pointCoordsZ_level.push_back( pointCoordZ );
            }
        }
    }
}

void PointProbe::calculateQuantities(SPtr<ProbeStruct> probeStruct, uint t, int level)
{
    vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(para->getParH(level)->numberofthreads, probeStruct->nPoints);
    int oldTimestepInTimeseries = this->outputTimeSeries ? calcOldTimestep(probeStruct->timestepInTimeseries, probeStruct->lastTimestepInOldTimeseries) : 0;
    int currentTimestep = this->outputTimeSeries ? probeStruct->timestepInTimeseries : 0;
    interpAndCalcQuantitiesKernel<<<grid.grid, grid.threads>>>(  probeStruct->pointIndicesD, probeStruct->nPoints, oldTimestepInTimeseries, currentTimestep, probeStruct->timestepInTimeAverage, probeStruct->nTimesteps,
                                                probeStruct->distXD, probeStruct->distYD, probeStruct->distZD,
                                                para->getParD(level)->velocityX, para->getParD(level)->velocityY, para->getParD(level)->velocityZ, para->getParD(level)->rho, 
                                                para->getParD(level)->neighborX, para->getParD(level)->neighborY, para->getParD(level)->neighborZ, 
                                                probeStruct->quantitiesD, probeStruct->arrayOffsetsD, probeStruct->quantitiesArrayD);
}

void PointProbe::addProbePoint(real pointCoordX, real pointCoordY, real pointCoordZ)
{
    this->pointCoordsX.push_back(pointCoordX);
    this->pointCoordsY.push_back(pointCoordY);
    this->pointCoordsZ.push_back(pointCoordZ);
}

void PointProbe::addProbePointsFromList(std::vector<real>& _pointCoordsX, std::vector<real>& _pointCoordsY, std::vector<real>& _pointCoordsZ)
{
    bool isSameLength = ( (_pointCoordsX.size()==_pointCoordsY.size()) && (_pointCoordsY.size()==_pointCoordsZ.size()));
    if (!isSameLength) throw std::runtime_error("Probe::addProbePointsFromList(): point lists have different lengths!");
    this->pointCoordsX.insert(this->pointCoordsX.end(), _pointCoordsX.begin(),  _pointCoordsX.end());
    this->pointCoordsY.insert(this->pointCoordsY.end(), _pointCoordsY.begin(),  _pointCoordsY.end());
    this->pointCoordsZ.insert(this->pointCoordsZ.end(), _pointCoordsZ.begin(),  _pointCoordsZ.end());
    printf("Added list of %u  points \n", uint(_pointCoordsX.size()) );
}

void PointProbe::getTaggedFluidNodes(GridProvider* gridProvider)
{
    for(int level=0; level<=para->getMaxLevel(); level++)
    {
        SPtr<ProbeStruct> probeStruct = this->getProbeStruct(level);
        std::vector<uint> probeIndices( probeStruct->pointIndicesH, probeStruct->pointIndicesH+probeStruct->nIndices);
        gridProvider->tagFluidNodeIndices( probeIndices, CollisionTemplate::WriteMacroVars, level);
    }
}

//! \}
