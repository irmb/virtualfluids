#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \author Martin Schoenherr
//=======================================================================================
#include "Calculation/Calculation.h" 
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;

__global__ void scaleFineToCoarseAdvectionDiffusion_Device(
    real* DC, 
    real* DF, 
    real* DD27C, 
    real* DD27F, 
    uint* neighborCX,
    uint* neighborCY,
    uint* neighborCZ,
    uint* neighborFX,
    uint* neighborFY,
    uint* neighborFZ,
    unsigned long long numberOfLBnodesCoarse, 
    unsigned long long numberOfLBnodesFine, 
    bool isEvenTimestep,
    uint* posC, 
    uint* posFSWB, 
    uint kFC, 
    real nu,
    real diffusivity_coarse,
    ICellNeigh offFC)
{
   real *feF, *fwF, *fnF, *fsF, *ftF, *fbF, 
       *fneF, *fswF, *fseF, *fnwF, *fteF, *fbwF, 
       *fbeF, *ftwF, *ftnF, *fbsF, *fbnF, *ftsF,
       *ftneF, *ftswF, *ftseF, *ftnwF, 
       *fbneF, *fbswF, *fbseF, *fbnwF;

   feF    = &DF[dP00 * numberOfLBnodesFine];
   fwF    = &DF[dM00 * numberOfLBnodesFine];
   fnF    = &DF[d0P0 * numberOfLBnodesFine];
   fsF    = &DF[d0M0 * numberOfLBnodesFine];
   ftF    = &DF[d00P * numberOfLBnodesFine];
   fbF    = &DF[d00M * numberOfLBnodesFine];
   fneF   = &DF[dPP0 * numberOfLBnodesFine];
   fswF   = &DF[dMM0 * numberOfLBnodesFine];
   fseF   = &DF[dPM0 * numberOfLBnodesFine];
   fnwF   = &DF[dMP0 * numberOfLBnodesFine];
   fteF   = &DF[dP0P * numberOfLBnodesFine];
   fbwF   = &DF[dM0M * numberOfLBnodesFine];
   fbeF   = &DF[dP0M * numberOfLBnodesFine];
   ftwF   = &DF[dM0P * numberOfLBnodesFine];
   ftnF   = &DF[d0PP * numberOfLBnodesFine];
   fbsF   = &DF[d0MM * numberOfLBnodesFine];
   fbnF   = &DF[d0PM * numberOfLBnodesFine];
   ftsF   = &DF[d0MP * numberOfLBnodesFine];
   ftneF  = &DF[dPPP * numberOfLBnodesFine];
   ftswF  = &DF[dMMP * numberOfLBnodesFine];
   ftseF  = &DF[dPMP * numberOfLBnodesFine];
   ftnwF  = &DF[dMPP * numberOfLBnodesFine];
   fbneF  = &DF[dPPM * numberOfLBnodesFine];
   fbswF  = &DF[dMMM * numberOfLBnodesFine];
   fbseF  = &DF[dPMM * numberOfLBnodesFine];
   fbnwF  = &DF[dMPM * numberOfLBnodesFine];

   real *feC, *fwC, *fnC, *fsC, *ftC, *fbC, 
       *fneC, *fswC, *fseC, *fnwC, *fteC, *fbwC, 
       *fbeC, *ftwC, *ftnC, *fbsC, *fbnC, *ftsC, 
       *ftneC, *ftswC, *ftseC, *ftnwC, 
       *fbneC, *fbswC, *fbseC, *fbnwC;

   if (isEvenTimestep)
   {
      feC    = &DC[dP00 * numberOfLBnodesCoarse];
      fwC    = &DC[dM00 * numberOfLBnodesCoarse];
      fnC    = &DC[d0P0 * numberOfLBnodesCoarse];
      fsC    = &DC[d0M0 * numberOfLBnodesCoarse];
      ftC    = &DC[d00P * numberOfLBnodesCoarse];
      fbC    = &DC[d00M * numberOfLBnodesCoarse];
      fneC   = &DC[dPP0 * numberOfLBnodesCoarse];
      fswC   = &DC[dMM0 * numberOfLBnodesCoarse];
      fseC   = &DC[dPM0 * numberOfLBnodesCoarse];
      fnwC   = &DC[dMP0 * numberOfLBnodesCoarse];
      fteC   = &DC[dP0P * numberOfLBnodesCoarse];
      fbwC   = &DC[dM0M * numberOfLBnodesCoarse];
      fbeC   = &DC[dP0M * numberOfLBnodesCoarse];
      ftwC   = &DC[dM0P * numberOfLBnodesCoarse];
      ftnC   = &DC[d0PP * numberOfLBnodesCoarse];
      fbsC   = &DC[d0MM * numberOfLBnodesCoarse];
      fbnC   = &DC[d0PM * numberOfLBnodesCoarse];
      ftsC   = &DC[d0MP * numberOfLBnodesCoarse];
      ftneC  = &DC[dPPP * numberOfLBnodesCoarse];
      ftswC  = &DC[dMMP * numberOfLBnodesCoarse];
      ftseC  = &DC[dPMP * numberOfLBnodesCoarse];
      ftnwC  = &DC[dMPP * numberOfLBnodesCoarse];
      fbneC  = &DC[dPPM * numberOfLBnodesCoarse];
      fbswC  = &DC[dMMM * numberOfLBnodesCoarse];
      fbseC  = &DC[dPMM * numberOfLBnodesCoarse];
      fbnwC  = &DC[dMPM * numberOfLBnodesCoarse];
   } 
   else
   {
      fwC    = &DC[dP00 * numberOfLBnodesCoarse];
      feC    = &DC[dM00 * numberOfLBnodesCoarse];
      fsC    = &DC[d0P0 * numberOfLBnodesCoarse];
      fnC    = &DC[d0M0 * numberOfLBnodesCoarse];
      fbC    = &DC[d00P * numberOfLBnodesCoarse];
      ftC    = &DC[d00M * numberOfLBnodesCoarse];
      fswC   = &DC[dPP0 * numberOfLBnodesCoarse];
      fneC   = &DC[dMM0 * numberOfLBnodesCoarse];
      fnwC   = &DC[dPM0 * numberOfLBnodesCoarse];
      fseC   = &DC[dMP0 * numberOfLBnodesCoarse];
      fbwC   = &DC[dP0P * numberOfLBnodesCoarse];
      fteC   = &DC[dM0M * numberOfLBnodesCoarse];
      ftwC   = &DC[dP0M * numberOfLBnodesCoarse];
      fbeC   = &DC[dM0P * numberOfLBnodesCoarse];
      fbsC   = &DC[d0PP * numberOfLBnodesCoarse];
      ftnC   = &DC[d0MM * numberOfLBnodesCoarse];
      ftsC   = &DC[d0PM * numberOfLBnodesCoarse];
      fbnC   = &DC[d0MP * numberOfLBnodesCoarse];
      fbswC  = &DC[dPPP * numberOfLBnodesCoarse];
      fbneC  = &DC[dMMP * numberOfLBnodesCoarse];
      fbnwC  = &DC[dPMP * numberOfLBnodesCoarse];
      fbseC  = &DC[dMPP * numberOfLBnodesCoarse];
      ftswC  = &DC[dPPM * numberOfLBnodesCoarse];
      ftneC  = &DC[dMMM * numberOfLBnodesCoarse];
      ftnwC  = &DC[dPMM * numberOfLBnodesCoarse];
      ftseC  = &DC[dMPM * numberOfLBnodesCoarse];
   }

   Distributions27 D27F;
   D27F.f[dP00] = &DD27F[dP00 * numberOfLBnodesFine];
   D27F.f[dM00] = &DD27F[dM00 * numberOfLBnodesFine];
   D27F.f[d0P0] = &DD27F[d0P0 * numberOfLBnodesFine];
   D27F.f[d0M0] = &DD27F[d0M0 * numberOfLBnodesFine];
   D27F.f[d00P] = &DD27F[d00P * numberOfLBnodesFine];
   D27F.f[d00M] = &DD27F[d00M * numberOfLBnodesFine];
   D27F.f[dPP0] = &DD27F[dPP0 * numberOfLBnodesFine];
   D27F.f[dMM0] = &DD27F[dMM0 * numberOfLBnodesFine];
   D27F.f[dPM0] = &DD27F[dPM0 * numberOfLBnodesFine];
   D27F.f[dMP0] = &DD27F[dMP0 * numberOfLBnodesFine];
   D27F.f[dP0P] = &DD27F[dP0P * numberOfLBnodesFine];
   D27F.f[dM0M] = &DD27F[dM0M * numberOfLBnodesFine];
   D27F.f[dP0M] = &DD27F[dP0M * numberOfLBnodesFine];
   D27F.f[dM0P] = &DD27F[dM0P * numberOfLBnodesFine];
   D27F.f[d0PP] = &DD27F[d0PP * numberOfLBnodesFine];
   D27F.f[d0MM] = &DD27F[d0MM * numberOfLBnodesFine];
   D27F.f[d0PM] = &DD27F[d0PM * numberOfLBnodesFine];
   D27F.f[d0MP] = &DD27F[d0MP * numberOfLBnodesFine];
   D27F.f[d000] = &DD27F[d000 * numberOfLBnodesFine];
   D27F.f[dPPP] = &DD27F[dPPP * numberOfLBnodesFine];
   D27F.f[dMMP] = &DD27F[dMMP * numberOfLBnodesFine];
   D27F.f[dPMP] = &DD27F[dPMP * numberOfLBnodesFine];
   D27F.f[dMPP] = &DD27F[dMPP * numberOfLBnodesFine];
   D27F.f[dPPM] = &DD27F[dPPM * numberOfLBnodesFine];
   D27F.f[dMMM] = &DD27F[dMMM * numberOfLBnodesFine];
   D27F.f[dPMM] = &DD27F[dPMM * numberOfLBnodesFine];
   D27F.f[dMPM] = &DD27F[dMPM * numberOfLBnodesFine];

   Distributions27 D27C;
   if (isEvenTimestep)
   {
      D27C.f[dP00] = &DD27C[dP00 * numberOfLBnodesCoarse];
      D27C.f[dM00] = &DD27C[dM00 * numberOfLBnodesCoarse];
      D27C.f[d0P0] = &DD27C[d0P0 * numberOfLBnodesCoarse];
      D27C.f[d0M0] = &DD27C[d0M0 * numberOfLBnodesCoarse];
      D27C.f[d00P] = &DD27C[d00P * numberOfLBnodesCoarse];
      D27C.f[d00M] = &DD27C[d00M * numberOfLBnodesCoarse];
      D27C.f[dPP0] = &DD27C[dPP0 * numberOfLBnodesCoarse];
      D27C.f[dMM0] = &DD27C[dMM0 * numberOfLBnodesCoarse];
      D27C.f[dPM0] = &DD27C[dPM0 * numberOfLBnodesCoarse];
      D27C.f[dMP0] = &DD27C[dMP0 * numberOfLBnodesCoarse];
      D27C.f[dP0P] = &DD27C[dP0P * numberOfLBnodesCoarse];
      D27C.f[dM0M] = &DD27C[dM0M * numberOfLBnodesCoarse];
      D27C.f[dP0M] = &DD27C[dP0M * numberOfLBnodesCoarse];
      D27C.f[dM0P] = &DD27C[dM0P * numberOfLBnodesCoarse];
      D27C.f[d0PP] = &DD27C[d0PP * numberOfLBnodesCoarse];
      D27C.f[d0MM] = &DD27C[d0MM * numberOfLBnodesCoarse];
      D27C.f[d0PM] = &DD27C[d0PM * numberOfLBnodesCoarse];
      D27C.f[d0MP] = &DD27C[d0MP * numberOfLBnodesCoarse];
      D27C.f[d000] = &DD27C[d000 * numberOfLBnodesCoarse];
      D27C.f[dPPP] = &DD27C[dPPP * numberOfLBnodesCoarse];
      D27C.f[dMMP] = &DD27C[dMMP * numberOfLBnodesCoarse];
      D27C.f[dPMP] = &DD27C[dPMP * numberOfLBnodesCoarse];
      D27C.f[dMPP] = &DD27C[dMPP * numberOfLBnodesCoarse];
      D27C.f[dPPM] = &DD27C[dPPM * numberOfLBnodesCoarse];
      D27C.f[dMMM] = &DD27C[dMMM * numberOfLBnodesCoarse];
      D27C.f[dPMM] = &DD27C[dPMM * numberOfLBnodesCoarse];
      D27C.f[dMPM] = &DD27C[dMPM * numberOfLBnodesCoarse];
   }
   else
   {
      D27C.f[dM00] = &DD27C[dP00 * numberOfLBnodesCoarse];
      D27C.f[dP00] = &DD27C[dM00 * numberOfLBnodesCoarse];
      D27C.f[d0M0] = &DD27C[d0P0 * numberOfLBnodesCoarse];
      D27C.f[d0P0] = &DD27C[d0M0 * numberOfLBnodesCoarse];
      D27C.f[d00M] = &DD27C[d00P * numberOfLBnodesCoarse];
      D27C.f[d00P] = &DD27C[d00M * numberOfLBnodesCoarse];
      D27C.f[dMM0] = &DD27C[dPP0 * numberOfLBnodesCoarse];
      D27C.f[dPP0] = &DD27C[dMM0 * numberOfLBnodesCoarse];
      D27C.f[dMP0] = &DD27C[dPM0 * numberOfLBnodesCoarse];
      D27C.f[dPM0] = &DD27C[dMP0 * numberOfLBnodesCoarse];
      D27C.f[dM0M] = &DD27C[dP0P * numberOfLBnodesCoarse];
      D27C.f[dP0P] = &DD27C[dM0M * numberOfLBnodesCoarse];
      D27C.f[dM0P] = &DD27C[dP0M * numberOfLBnodesCoarse];
      D27C.f[dP0M] = &DD27C[dM0P * numberOfLBnodesCoarse];
      D27C.f[d0MM] = &DD27C[d0PP * numberOfLBnodesCoarse];
      D27C.f[d0PP] = &DD27C[d0MM * numberOfLBnodesCoarse];
      D27C.f[d0MP] = &DD27C[d0PM * numberOfLBnodesCoarse];
      D27C.f[d0PM] = &DD27C[d0MP * numberOfLBnodesCoarse];
      D27C.f[d000] = &DD27C[d000 * numberOfLBnodesCoarse];
      D27C.f[dMMM] = &DD27C[dPPP * numberOfLBnodesCoarse];
      D27C.f[dPPM] = &DD27C[dMMP * numberOfLBnodesCoarse];
      D27C.f[dMPM] = &DD27C[dPMP * numberOfLBnodesCoarse];
      D27C.f[dPMM] = &DD27C[dMPP * numberOfLBnodesCoarse];
      D27C.f[dMMP] = &DD27C[dPPM * numberOfLBnodesCoarse];
      D27C.f[dPPP] = &DD27C[dMMM * numberOfLBnodesCoarse];
      D27C.f[dMPP] = &DD27C[dPMM * numberOfLBnodesCoarse];
      D27C.f[dPMP] = &DD27C[dMPM * numberOfLBnodesCoarse];
   }

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  ix = threadIdx.x;
   const unsigned  iy = blockIdx.x; 
   const unsigned  iz = blockIdx.y; 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*iz + iy) + ix;

   ////////////////////////////////////////////////////////////////////////////////
   real vx1,vx2,vx3,cu_sq;
   real f_E,f_W,f_N,f_S,f_T,f_B,f_NE,f_SW,f_SE,f_NW,f_TE,f_BW,f_BE,f_TW,f_TN,f_BS,f_BN,f_TS,f_TNE,f_TSW,f_TSE,f_TNW,f_BNE,f_BSW,f_BSE,f_BNW;

   real f27E,f27W,f27N,f27S,f27T,f27B,f27NE,f27SW,f27SE,f27NW,f27TE,f27BW,f27BE,f27TW,f27TN,f27BS,f27BN,f27TS,f27ZERO,f27TNE,f27TSW,f27TSE,f27TNW,f27BNE,f27BSW,f27BSE,f27BNW;
   real Mx,My,Mz; 
   real Conc_F_SWB, Conc_F_SWT, Conc_F_SET, Conc_F_SEB, Conc_F_NWB, Conc_F_NWT, Conc_F_NET, Conc_F_NEB;

   real omegaD_C = c2o1 / (c6o1 * diffusivity_coarse + c1o1);
   real omegaD_F = c2o1 / (c6o1 * diffusivity_coarse*c2o1 + c1o1);

   real xoff,    yoff,    zoff;
   real xoff_sq, yoff_sq, zoff_sq;

   if(k<kFC){
      //////////////////////////////////////////////////////////////////////////
      xoff    = offFC.x[k];
      yoff    = offFC.y[k];
      zoff    = offFC.z[k];
      xoff_sq = xoff * xoff;
      yoff_sq = yoff * yoff;
      zoff_sq = zoff * zoff;
      //////////////////////////////////////////////////////////////////////////
      //SWB//
      //////////////////////////////////////////////////////////////////////////
      //index 0
      unsigned int k0zero= posFSWB[k];
      unsigned int k0w   = neighborFX[k0zero];
      unsigned int k0s   = neighborFY[k0zero];
      unsigned int k0b   = neighborFZ[k0zero];
      unsigned int k0sw  = neighborFY[k0w];
      unsigned int k0bw  = neighborFZ[k0w];
      unsigned int k0bs  = neighborFZ[k0s];
      unsigned int k0bsw = neighborFZ[k0sw];
      //////////////////////////////////////////////////////////////////////////
      //index 
      unsigned int kzero= k0zero;
      unsigned int kw   = k0w;   
      unsigned int ks   = k0s;   
      unsigned int kb   = k0b;   
      unsigned int ksw  = k0sw;  
      unsigned int kbw  = k0bw;  
      unsigned int kbs  = k0bs;  
      unsigned int kbsw = k0bsw; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feF[kzero];
      f_W    = fwF[kw];
      f_N    = fnF[kzero];
      f_S    = fsF[ks];
      f_T    = ftF[kzero];
      f_B    = fbF[kb];
      f_NE   = fneF[kzero];
      f_SW   = fswF[ksw];
      f_SE   = fseF[ks];
      f_NW   = fnwF[kw];
      f_TE   = fteF[kzero];
      f_BW   = fbwF[kbw];
      f_BE   = fbeF[kb];
      f_TW   = ftwF[kw];
      f_TN   = ftnF[kzero];
      f_BS   = fbsF[kbs];
      f_BN   = fbnF[kb];
      f_TS   = ftsF[ks];
      f_TNE  = ftneF[kzero];
      f_TSW  = ftswF[ksw];
      f_TSE  = ftseF[ks];
      f_TNW  = ftnwF[kw];
      f_BNE  = fbneF[kb];
      f_BSW  = fbswF[kbsw];
      f_BSE  = fbseF[kbs];
      f_BNW  = fbnwF[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27F.f[dP00])[kzero];//ke
      f27W    =  (D27F.f[dM00])[kw   ];
      f27N    =  (D27F.f[d0P0])[kzero];//kn
      f27S    =  (D27F.f[d0M0])[ks   ];
      f27T    =  (D27F.f[d00P])[kzero];//kt
      f27B    =  (D27F.f[d00M])[kb   ];
      f27NE   =  (D27F.f[dPP0])[kzero];//kne
      f27SW   =  (D27F.f[dMM0])[ksw  ];
      f27SE   =  (D27F.f[dPM0])[ks   ];//kse
      f27NW   =  (D27F.f[dMP0])[kw   ];//knw
      f27TE   =  (D27F.f[dP0P])[kzero];//kte
      f27BW   =  (D27F.f[dM0M])[kbw  ];
      f27BE   =  (D27F.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27F.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27F.f[d0PP])[kzero];//ktn
      f27BS   =  (D27F.f[d0MM])[kbs  ];
      f27BN   =  (D27F.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27F.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27F.f[d000])[kzero];//kzero
      f27TNE   = (D27F.f[dPPP])[kzero];//ktne
      f27TSW   = (D27F.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27F.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27F.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27F.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27F.f[dMMM])[kbsw ];
      f27BSE   = (D27F.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27F.f[dMPM])[kbw  ];//kbnw

      Conc_F_SWB = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW;

      vx1  = f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  = f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  = f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_FSWB = (Conc_F_SWB * vx1 - Mx) * (c3o1*omegaD_F);
      real Diff_Conc_Y_FSWB = (Conc_F_SWB * vx2 - My) * (c3o1*omegaD_F);
      real Diff_Conc_Z_FSWB = (Conc_F_SWB * vx3 - Mz) * (c3o1*omegaD_F);




      //////////////////////////////////////////////////////////////////////////
      //SWT//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kb;
      kw   = kbw;   
      ks   = kbs;   
      kb   = neighborFZ[kb];   
      ksw  = kbsw;  
      kbw  = neighborFZ[kbw];  
      kbs  = neighborFZ[kbs];  
      kbsw = neighborFZ[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feF[kzero];
      f_W    = fwF[kw];
      f_N    = fnF[kzero];
      f_S    = fsF[ks];
      f_T    = ftF[kzero];
      f_B    = fbF[kb];
      f_NE   = fneF[kzero];
      f_SW   = fswF[ksw];
      f_SE   = fseF[ks];
      f_NW   = fnwF[kw];
      f_TE   = fteF[kzero];
      f_BW   = fbwF[kbw];
      f_BE   = fbeF[kb];
      f_TW   = ftwF[kw];
      f_TN   = ftnF[kzero];
      f_BS   = fbsF[kbs];
      f_BN   = fbnF[kb];
      f_TS   = ftsF[ks];
      f_TNE  = ftneF[kzero];
      f_TSW  = ftswF[ksw];
      f_TSE  = ftseF[ks];
      f_TNW  = ftnwF[kw];
      f_BNE  = fbneF[kb];
      f_BSW  = fbswF[kbsw];
      f_BSE  = fbseF[kbs];
      f_BNW  = fbnwF[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27F.f[dP00])[kzero];//ke
      f27W    =  (D27F.f[dM00])[kw   ];
      f27N    =  (D27F.f[d0P0])[kzero];//kn
      f27S    =  (D27F.f[d0M0])[ks   ];
      f27T    =  (D27F.f[d00P])[kzero];//kt
      f27B    =  (D27F.f[d00M])[kb   ];
      f27NE   =  (D27F.f[dPP0])[kzero];//kne
      f27SW   =  (D27F.f[dMM0])[ksw  ];
      f27SE   =  (D27F.f[dPM0])[ks   ];//kse
      f27NW   =  (D27F.f[dMP0])[kw   ];//knw
      f27TE   =  (D27F.f[dP0P])[kzero];//kte
      f27BW   =  (D27F.f[dM0M])[kbw  ];
      f27BE   =  (D27F.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27F.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27F.f[d0PP])[kzero];//ktn
      f27BS   =  (D27F.f[d0MM])[kbs  ];
      f27BN   =  (D27F.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27F.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27F.f[d000])[kzero];//kzero
      f27TNE   = (D27F.f[dPPP])[kzero];//ktne
      f27TSW   = (D27F.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27F.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27F.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27F.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27F.f[dMMM])[kbsw ];
      f27BSE   = (D27F.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27F.f[dMPM])[kbw  ];//kbnw

      Conc_F_SWT = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW;

      vx1  = f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  = f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  = f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_FSWT = (Conc_F_SWT * vx1 - Mx) * (c3o1*omegaD_F);
      real Diff_Conc_Y_FSWT = (Conc_F_SWT * vx2 - My) * (c3o1*omegaD_F);
      real Diff_Conc_Z_FSWT = (Conc_F_SWT * vx3 - Mz) * (c3o1*omegaD_F);




      //////////////////////////////////////////////////////////////////////////
      //SET//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kw;
      kw   = neighborFX[kw];   
      ks   = ksw;   
      kb   = kbw;   
      ksw  = neighborFX[ksw];  
      kbw  = neighborFX[kbw];  
      kbs  = kbsw;  
      kbsw = neighborFX[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feF[kzero];
      f_W    = fwF[kw];
      f_N    = fnF[kzero];
      f_S    = fsF[ks];
      f_T    = ftF[kzero];
      f_B    = fbF[kb];
      f_NE   = fneF[kzero];
      f_SW   = fswF[ksw];
      f_SE   = fseF[ks];
      f_NW   = fnwF[kw];
      f_TE   = fteF[kzero];
      f_BW   = fbwF[kbw];
      f_BE   = fbeF[kb];
      f_TW   = ftwF[kw];
      f_TN   = ftnF[kzero];
      f_BS   = fbsF[kbs];
      f_BN   = fbnF[kb];
      f_TS   = ftsF[ks];
      f_TNE  = ftneF[kzero];
      f_TSW  = ftswF[ksw];
      f_TSE  = ftseF[ks];
      f_TNW  = ftnwF[kw];
      f_BNE  = fbneF[kb];
      f_BSW  = fbswF[kbsw];
      f_BSE  = fbseF[kbs];
      f_BNW  = fbnwF[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27F.f[dP00])[kzero];//ke
      f27W    =  (D27F.f[dM00])[kw   ];
      f27N    =  (D27F.f[d0P0])[kzero];//kn
      f27S    =  (D27F.f[d0M0])[ks   ];
      f27T    =  (D27F.f[d00P])[kzero];//kt
      f27B    =  (D27F.f[d00M])[kb   ];
      f27NE   =  (D27F.f[dPP0])[kzero];//kne
      f27SW   =  (D27F.f[dMM0])[ksw  ];
      f27SE   =  (D27F.f[dPM0])[ks   ];//kse
      f27NW   =  (D27F.f[dMP0])[kw   ];//knw
      f27TE   =  (D27F.f[dP0P])[kzero];//kte
      f27BW   =  (D27F.f[dM0M])[kbw  ];
      f27BE   =  (D27F.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27F.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27F.f[d0PP])[kzero];//ktn
      f27BS   =  (D27F.f[d0MM])[kbs  ];
      f27BN   =  (D27F.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27F.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27F.f[d000])[kzero];//kzero
      f27TNE   = (D27F.f[dPPP])[kzero];//ktne
      f27TSW   = (D27F.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27F.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27F.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27F.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27F.f[dMMM])[kbsw ];
      f27BSE   = (D27F.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27F.f[dMPM])[kbw  ];//kbnw

      Conc_F_SET = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW;

      vx1  = f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  = f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  = f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_FSET = (Conc_F_SET * vx1 - Mx) * (c3o1*omegaD_F);
      real Diff_Conc_Y_FSET = (Conc_F_SET * vx2 - My) * (c3o1*omegaD_F);
      real Diff_Conc_Z_FSET = (Conc_F_SET * vx3 - Mz) * (c3o1*omegaD_F);




      //////////////////////////////////////////////////////////////////////////
      //SEB//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kb   = kzero;   
      kbw  = kw;  
      kbs  = ks;  
      kbsw = ksw; 
      kzero= k0w;
      kw   = neighborFX[k0w];   
      ks   = k0sw;   
      ksw  = neighborFX[k0sw];  
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feF[kzero];
      f_W    = fwF[kw];
      f_N    = fnF[kzero];
      f_S    = fsF[ks];
      f_T    = ftF[kzero];
      f_B    = fbF[kb];
      f_NE   = fneF[kzero];
      f_SW   = fswF[ksw];
      f_SE   = fseF[ks];
      f_NW   = fnwF[kw];
      f_TE   = fteF[kzero];
      f_BW   = fbwF[kbw];
      f_BE   = fbeF[kb];
      f_TW   = ftwF[kw];
      f_TN   = ftnF[kzero];
      f_BS   = fbsF[kbs];
      f_BN   = fbnF[kb];
      f_TS   = ftsF[ks];
      f_TNE  = ftneF[kzero];
      f_TSW  = ftswF[ksw];
      f_TSE  = ftseF[ks];
      f_TNW  = ftnwF[kw];
      f_BNE  = fbneF[kb];
      f_BSW  = fbswF[kbsw];
      f_BSE  = fbseF[kbs];
      f_BNW  = fbnwF[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27F.f[dP00])[kzero];//ke
      f27W    =  (D27F.f[dM00])[kw   ];
      f27N    =  (D27F.f[d0P0])[kzero];//kn
      f27S    =  (D27F.f[d0M0])[ks   ];
      f27T    =  (D27F.f[d00P])[kzero];//kt
      f27B    =  (D27F.f[d00M])[kb   ];
      f27NE   =  (D27F.f[dPP0])[kzero];//kne
      f27SW   =  (D27F.f[dMM0])[ksw  ];
      f27SE   =  (D27F.f[dPM0])[ks   ];//kse
      f27NW   =  (D27F.f[dMP0])[kw   ];//knw
      f27TE   =  (D27F.f[dP0P])[kzero];//kte
      f27BW   =  (D27F.f[dM0M])[kbw  ];
      f27BE   =  (D27F.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27F.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27F.f[d0PP])[kzero];//ktn
      f27BS   =  (D27F.f[d0MM])[kbs  ];
      f27BN   =  (D27F.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27F.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27F.f[d000])[kzero];//kzero
      f27TNE   = (D27F.f[dPPP])[kzero];//ktne
      f27TSW   = (D27F.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27F.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27F.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27F.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27F.f[dMMM])[kbsw ];
      f27BSE   = (D27F.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27F.f[dMPM])[kbw  ];//kbnw

      Conc_F_SEB = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW;

      vx1  = f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  = f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  = f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_FSEB = (Conc_F_SEB * vx1 - Mx) * (c3o1*omegaD_F);
      real Diff_Conc_Y_FSEB = (Conc_F_SEB * vx2 - My) * (c3o1*omegaD_F);
      real Diff_Conc_Z_FSEB = (Conc_F_SEB * vx3 - Mz) * (c3o1*omegaD_F);




      //////////////////////////////////////////////////////////////////////////
      //NWB//
      //////////////////////////////////////////////////////////////////////////
      //index 0
      k0zero= k0s;
      k0w   = k0sw;
      k0s   = neighborFY[k0s];
      k0b   = k0bs;
      k0sw  = neighborFY[k0sw];
      k0bw  = k0bsw;
      k0bs  = neighborFY[k0bs];
      k0bsw = neighborFY[k0bsw];
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= k0zero;
      kw   = k0w;   
      ks   = k0s;   
      kb   = k0b;   
      ksw  = k0sw;  
      kbw  = k0bw;  
      kbs  = k0bs;  
      kbsw = k0bsw; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feF[kzero];
      f_W    = fwF[kw];
      f_N    = fnF[kzero];
      f_S    = fsF[ks];
      f_T    = ftF[kzero];
      f_B    = fbF[kb];
      f_NE   = fneF[kzero];
      f_SW   = fswF[ksw];
      f_SE   = fseF[ks];
      f_NW   = fnwF[kw];
      f_TE   = fteF[kzero];
      f_BW   = fbwF[kbw];
      f_BE   = fbeF[kb];
      f_TW   = ftwF[kw];
      f_TN   = ftnF[kzero];
      f_BS   = fbsF[kbs];
      f_BN   = fbnF[kb];
      f_TS   = ftsF[ks];
      f_TNE  = ftneF[kzero];
      f_TSW  = ftswF[ksw];
      f_TSE  = ftseF[ks];
      f_TNW  = ftnwF[kw];
      f_BNE  = fbneF[kb];
      f_BSW  = fbswF[kbsw];
      f_BSE  = fbseF[kbs];
      f_BNW  = fbnwF[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27F.f[dP00])[kzero];//ke
      f27W    =  (D27F.f[dM00])[kw   ];
      f27N    =  (D27F.f[d0P0])[kzero];//kn
      f27S    =  (D27F.f[d0M0])[ks   ];
      f27T    =  (D27F.f[d00P])[kzero];//kt
      f27B    =  (D27F.f[d00M])[kb   ];
      f27NE   =  (D27F.f[dPP0])[kzero];//kne
      f27SW   =  (D27F.f[dMM0])[ksw  ];
      f27SE   =  (D27F.f[dPM0])[ks   ];//kse
      f27NW   =  (D27F.f[dMP0])[kw   ];//knw
      f27TE   =  (D27F.f[dP0P])[kzero];//kte
      f27BW   =  (D27F.f[dM0M])[kbw  ];
      f27BE   =  (D27F.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27F.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27F.f[d0PP])[kzero];//ktn
      f27BS   =  (D27F.f[d0MM])[kbs  ];
      f27BN   =  (D27F.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27F.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27F.f[d000])[kzero];//kzero
      f27TNE   = (D27F.f[dPPP])[kzero];//ktne
      f27TSW   = (D27F.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27F.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27F.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27F.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27F.f[dMMM])[kbsw ];
      f27BSE   = (D27F.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27F.f[dMPM])[kbw  ];//kbnw

      Conc_F_NWB = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW;

      vx1  = f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  = f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  = f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_FNWB = (Conc_F_NWB * vx1 - Mx) * (c3o1*omegaD_F);
      real Diff_Conc_Y_FNWB = (Conc_F_NWB * vx2 - My) * (c3o1*omegaD_F);
      real Diff_Conc_Z_FNWB = (Conc_F_NWB * vx3 - Mz) * (c3o1*omegaD_F);




      //////////////////////////////////////////////////////////////////////////
      //NWT//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kb;
      kw   = kbw;   
      ks   = kbs;   
      kb   = neighborFZ[kb];   
      ksw  = kbsw;  
      kbw  = neighborFZ[kbw];  
      kbs  = neighborFZ[kbs];  
      kbsw = neighborFZ[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feF[kzero];
      f_W    = fwF[kw];
      f_N    = fnF[kzero];
      f_S    = fsF[ks];
      f_T    = ftF[kzero];
      f_B    = fbF[kb];
      f_NE   = fneF[kzero];
      f_SW   = fswF[ksw];
      f_SE   = fseF[ks];
      f_NW   = fnwF[kw];
      f_TE   = fteF[kzero];
      f_BW   = fbwF[kbw];
      f_BE   = fbeF[kb];
      f_TW   = ftwF[kw];
      f_TN   = ftnF[kzero];
      f_BS   = fbsF[kbs];
      f_BN   = fbnF[kb];
      f_TS   = ftsF[ks];
      f_TNE  = ftneF[kzero];
      f_TSW  = ftswF[ksw];
      f_TSE  = ftseF[ks];
      f_TNW  = ftnwF[kw];
      f_BNE  = fbneF[kb];
      f_BSW  = fbswF[kbsw];
      f_BSE  = fbseF[kbs];
      f_BNW  = fbnwF[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27F.f[dP00])[kzero];//ke
      f27W    =  (D27F.f[dM00])[kw   ];
      f27N    =  (D27F.f[d0P0])[kzero];//kn
      f27S    =  (D27F.f[d0M0])[ks   ];
      f27T    =  (D27F.f[d00P])[kzero];//kt
      f27B    =  (D27F.f[d00M])[kb   ];
      f27NE   =  (D27F.f[dPP0])[kzero];//kne
      f27SW   =  (D27F.f[dMM0])[ksw  ];
      f27SE   =  (D27F.f[dPM0])[ks   ];//kse
      f27NW   =  (D27F.f[dMP0])[kw   ];//knw
      f27TE   =  (D27F.f[dP0P])[kzero];//kte
      f27BW   =  (D27F.f[dM0M])[kbw  ];
      f27BE   =  (D27F.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27F.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27F.f[d0PP])[kzero];//ktn
      f27BS   =  (D27F.f[d0MM])[kbs  ];
      f27BN   =  (D27F.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27F.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27F.f[d000])[kzero];//kzero
      f27TNE   = (D27F.f[dPPP])[kzero];//ktne
      f27TSW   = (D27F.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27F.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27F.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27F.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27F.f[dMMM])[kbsw ];
      f27BSE   = (D27F.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27F.f[dMPM])[kbw  ];//kbnw

      Conc_F_NWT = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW;

      vx1  = f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  = f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  = f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_FNWT = (Conc_F_NWT * vx1 - Mx) * (c3o1*omegaD_F);
      real Diff_Conc_Y_FNWT = (Conc_F_NWT * vx2 - My) * (c3o1*omegaD_F);
      real Diff_Conc_Z_FNWT = (Conc_F_NWT * vx3 - Mz) * (c3o1*omegaD_F);




      //////////////////////////////////////////////////////////////////////////
      //NET//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kw;
      kw   = neighborFX[kw];   
      ks   = ksw;   
      kb   = kbw;   
      ksw  = neighborFX[ksw];  
      kbw  = neighborFX[kbw];  
      kbs  = kbsw;  
      kbsw = neighborFX[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feF[kzero];
      f_W    = fwF[kw];
      f_N    = fnF[kzero];
      f_S    = fsF[ks];
      f_T    = ftF[kzero];
      f_B    = fbF[kb];
      f_NE   = fneF[kzero];
      f_SW   = fswF[ksw];
      f_SE   = fseF[ks];
      f_NW   = fnwF[kw];
      f_TE   = fteF[kzero];
      f_BW   = fbwF[kbw];
      f_BE   = fbeF[kb];
      f_TW   = ftwF[kw];
      f_TN   = ftnF[kzero];
      f_BS   = fbsF[kbs];
      f_BN   = fbnF[kb];
      f_TS   = ftsF[ks];
      f_TNE  = ftneF[kzero];
      f_TSW  = ftswF[ksw];
      f_TSE  = ftseF[ks];
      f_TNW  = ftnwF[kw];
      f_BNE  = fbneF[kb];
      f_BSW  = fbswF[kbsw];
      f_BSE  = fbseF[kbs];
      f_BNW  = fbnwF[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27F.f[dP00])[kzero];//ke
      f27W    =  (D27F.f[dM00])[kw   ];
      f27N    =  (D27F.f[d0P0])[kzero];//kn
      f27S    =  (D27F.f[d0M0])[ks   ];
      f27T    =  (D27F.f[d00P])[kzero];//kt
      f27B    =  (D27F.f[d00M])[kb   ];
      f27NE   =  (D27F.f[dPP0])[kzero];//kne
      f27SW   =  (D27F.f[dMM0])[ksw  ];
      f27SE   =  (D27F.f[dPM0])[ks   ];//kse
      f27NW   =  (D27F.f[dMP0])[kw   ];//knw
      f27TE   =  (D27F.f[dP0P])[kzero];//kte
      f27BW   =  (D27F.f[dM0M])[kbw  ];
      f27BE   =  (D27F.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27F.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27F.f[d0PP])[kzero];//ktn
      f27BS   =  (D27F.f[d0MM])[kbs  ];
      f27BN   =  (D27F.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27F.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27F.f[d000])[kzero];//kzero
      f27TNE   = (D27F.f[dPPP])[kzero];//ktne
      f27TSW   = (D27F.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27F.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27F.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27F.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27F.f[dMMM])[kbsw ];
      f27BSE   = (D27F.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27F.f[dMPM])[kbw  ];//kbnw

      Conc_F_NET = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW;

      vx1  = f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  = f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  = f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_FNET = (Conc_F_NET * vx1 - Mx) * (c3o1*omegaD_F);
      real Diff_Conc_Y_FNET = (Conc_F_NET * vx2 - My) * (c3o1*omegaD_F);
      real Diff_Conc_Z_FNET = (Conc_F_NET * vx3 - Mz) * (c3o1*omegaD_F);




      //////////////////////////////////////////////////////////////////////////
      //NEB//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kb   = kzero;   
      kbw  = kw;  
      kbs  = ks;  
      kbsw = ksw; 
      kzero= k0w;
      kw   = neighborFX[k0w];   
      ks   = k0sw;   
      ksw  = neighborFX[k0sw];  
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feF[kzero];
      f_W    = fwF[kw];
      f_N    = fnF[kzero];
      f_S    = fsF[ks];
      f_T    = ftF[kzero];
      f_B    = fbF[kb];
      f_NE   = fneF[kzero];
      f_SW   = fswF[ksw];
      f_SE   = fseF[ks];
      f_NW   = fnwF[kw];
      f_TE   = fteF[kzero];
      f_BW   = fbwF[kbw];
      f_BE   = fbeF[kb];
      f_TW   = ftwF[kw];
      f_TN   = ftnF[kzero];
      f_BS   = fbsF[kbs];
      f_BN   = fbnF[kb];
      f_TS   = ftsF[ks];
      f_TNE  = ftneF[kzero];
      f_TSW  = ftswF[ksw];
      f_TSE  = ftseF[ks];
      f_TNW  = ftnwF[kw];
      f_BNE  = fbneF[kb];
      f_BSW  = fbswF[kbsw];
      f_BSE  = fbseF[kbs];
      f_BNW  = fbnwF[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27F.f[dP00])[kzero];//ke
      f27W    =  (D27F.f[dM00])[kw   ];
      f27N    =  (D27F.f[d0P0])[kzero];//kn
      f27S    =  (D27F.f[d0M0])[ks   ];
      f27T    =  (D27F.f[d00P])[kzero];//kt
      f27B    =  (D27F.f[d00M])[kb   ];
      f27NE   =  (D27F.f[dPP0])[kzero];//kne
      f27SW   =  (D27F.f[dMM0])[ksw  ];
      f27SE   =  (D27F.f[dPM0])[ks   ];//kse
      f27NW   =  (D27F.f[dMP0])[kw   ];//knw
      f27TE   =  (D27F.f[dP0P])[kzero];//kte
      f27BW   =  (D27F.f[dM0M])[kbw  ];
      f27BE   =  (D27F.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27F.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27F.f[d0PP])[kzero];//ktn
      f27BS   =  (D27F.f[d0MM])[kbs  ];
      f27BN   =  (D27F.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27F.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27F.f[d000])[kzero];//kzero
      f27TNE   = (D27F.f[dPPP])[kzero];//ktne
      f27TSW   = (D27F.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27F.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27F.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27F.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27F.f[dMMM])[kbsw ];
      f27BSE   = (D27F.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27F.f[dMPM])[kbw  ];//kbnw

      Conc_F_NEB = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW;

      vx1  = f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  = f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  = f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_FNEB = (Conc_F_NEB * vx1 - Mx) * (c3o1*omegaD_F);
      real Diff_Conc_Y_FNEB = (Conc_F_NEB * vx2 - My) * (c3o1*omegaD_F);
      real Diff_Conc_Z_FNEB = (Conc_F_NEB * vx3 - Mz) * (c3o1*omegaD_F);




      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //linear
      //real Diff_Conc_XX = zero;
      //real Diff_Conc_YY = zero;
      //real Diff_Conc_ZZ = zero;
      //quadratic
      real Diff_Conc_XX = ((Diff_Conc_X_FNEB + Diff_Conc_X_FSEB + Diff_Conc_X_FNET + Diff_Conc_X_FSET) - (Diff_Conc_X_FNWB + Diff_Conc_X_FSWB + Diff_Conc_X_FNWT + Diff_Conc_X_FSWT)) * c1o4;
      real Diff_Conc_YY = ((Diff_Conc_Y_FNEB + Diff_Conc_Y_FNWB + Diff_Conc_Y_FNET + Diff_Conc_Y_FNWT) - (Diff_Conc_Y_FSEB + Diff_Conc_Y_FSWB + Diff_Conc_Y_FSET + Diff_Conc_Y_FSWT)) * c1o4;
      real Diff_Conc_ZZ = ((Diff_Conc_Z_FSET + Diff_Conc_Z_FSWT + Diff_Conc_Z_FNET + Diff_Conc_Z_FNWT) - (Diff_Conc_Z_FSEB + Diff_Conc_Z_FSWB + Diff_Conc_Z_FNEB + Diff_Conc_Z_FNWB)) * c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


      real dx = c1o4 * (Conc_F_NEB + Conc_F_NET - Conc_F_NWB - Conc_F_NWT + Conc_F_SEB + Conc_F_SET - Conc_F_SWB - Conc_F_SWT);
      real dy = c1o4 * (Conc_F_NEB + Conc_F_NET + Conc_F_NWB + Conc_F_NWT - Conc_F_SEB - Conc_F_SET - Conc_F_SWB - Conc_F_SWT);
      real dz = c1o4 * (-Conc_F_NEB + Conc_F_NET - Conc_F_NWB + Conc_F_NWT - Conc_F_SEB + Conc_F_SET - Conc_F_SWB + Conc_F_SWT);
      real dxx = Diff_Conc_XX * c1o2;
      real dyy = Diff_Conc_YY * c1o2;
      real dzz = Diff_Conc_ZZ * c1o2;
      real dxy = c1o2 * (Conc_F_NEB + Conc_F_NET - Conc_F_NWB - Conc_F_NWT - Conc_F_SEB - Conc_F_SET + Conc_F_SWB + Conc_F_SWT);
      real dyz = c1o2 * (-Conc_F_NEB + Conc_F_NET - Conc_F_NWB + Conc_F_NWT + Conc_F_SEB - Conc_F_SET + Conc_F_SWB - Conc_F_SWT);
      real dxz = c1o2 * (-Conc_F_NEB + Conc_F_NET + Conc_F_NWB - Conc_F_NWT - Conc_F_SEB + Conc_F_SET + Conc_F_SWB - Conc_F_SWT);

      real d0 = c1o8 * (-c2o1 * dxx - c2o1 * dyy - c2o1 * dzz + Conc_F_NEB + Conc_F_NET + Conc_F_NWB + Conc_F_NWT + Conc_F_SEB + Conc_F_SET + Conc_F_SWB + Conc_F_SWT);

      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //
      // offset for refinement into the wall
      //
      // X------X
      // |      |
      // |   ---+-->X     ----> off-vector
      // |      |
      // X------X
      //
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      d0 = d0 + xoff * dx + yoff * dy + zoff * dz + xoff_sq * dxx + yoff_sq * dyy + zoff_sq * dzz + xoff*yoff*dxy + xoff*zoff*dxz + yoff*zoff*dyz;
      dx = dx + c2o1 * xoff * dxx + yoff * dxy + zoff * dxz;
      dy = dy + c2o1 * yoff * dyy + xoff * dxy + zoff * dyz;
      dz = dz + c2o1 * zoff * dzz + xoff * dxz + yoff * dyz;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////




      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position C 0.5, 0.5, 0.5
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //x = zero;
      //y = zero;
      //z = zero;

      //////////////////////////////////////////////////////////////////////////
      //index 0
      kzero= posC[k];
      kw   = neighborCX[kzero];
      ks   = neighborCY[kzero];
      kb   = neighborCZ[kzero];
      ksw  = neighborCY[kw];
      kbw  = neighborCZ[kw];
      kbs  = neighborCZ[ks];
      kbsw = neighborCZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      vx1=feC[kzero]+fneC[kzero]+fseC[ks]+fteC[kzero]+fbeC[kb]-fwC[kw]-fnwC[kw]-fswC[ksw]-ftwC[kw]-fbwC[kbw]+ftneC[kzero]-ftswC[ksw]+ftseC[ks]-ftnwC[kw]+fbneC[kb]-fbswC[kbsw]+fbseC[kbs]-fbnwC[kbw];
      vx2=fnC[kzero]+fneC[kzero]+fnwC[kw]+ftnC[kzero]+fbnC[kb]-fsC[ks]-fseC[ks]-fswC[ksw]-ftsC[ks]-fbsC[kbs]+ftneC[kzero]-ftswC[ksw]-ftseC[ks]+ftnwC[kw]+fbneC[kb]-fbswC[kbsw]-fbseC[kbs]+fbnwC[kbw];
      vx3=ftC[kzero]+fteC[kzero]+ftwC[kw]+ftnC[kzero]+ftsC[ks]-fbC[kb]-fbeC[kb]-fbwC[kbw]-fbnC[kb]-fbsC[kbs]+ftneC[kzero]+ftswC[ksw]+ftseC[ks]+ftnwC[kw]-fbneC[kb]-fbswC[kbsw]-fbseC[kbs]-fbnwC[kbw];

      real Conc_C = d0;

      real Diff_Conc_X_C = dx;
      real Diff_Conc_Y_C = dy;
      real Diff_Conc_Z_C = dz;

      Mx = Conc_C*vx1-(c1o1)/(c3o1*omegaD_C)*c2o1*Diff_Conc_X_C;
      My = Conc_C*vx2-(c1o1)/(c3o1*omegaD_C)*c2o1*Diff_Conc_Y_C;
      Mz = Conc_C*vx3-(c1o1)/(c3o1*omegaD_C)*c2o1*Diff_Conc_Z_C;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27C.f[d000])[kzero] =   c8o27* Conc_C*(c1o1-cu_sq);
      (D27C.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_C*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27C.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_C*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27C.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_C*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27C.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_C*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27C.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_C*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27C.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_C*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27C.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_C*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27C.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_C*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27C.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_C*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27C.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_C*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27C.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_C*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27C.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_C*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27C.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_C*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27C.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_C*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27C.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_C*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27C.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_C*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27C.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_C*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27C.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_C*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27C.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_C*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27C.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_C*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27C.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_C*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27C.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_C*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27C.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_C*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27C.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_C*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27C.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_C*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27C.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_C*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));

   }
}

