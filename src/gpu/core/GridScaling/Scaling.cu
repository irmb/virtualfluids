#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \author Martin Schoenherr, Soeren Peters
//======================================================================================
#include "Scaling.cuh"

#include <hip/hip_runtime_api.h>
#include <cuda_helper/CudaGrid.h>

#include "lbm/MacroscopicQuantities.h"
#include "lbm/constants/D3Q27.h"

#include "basics/constants/NumericConstants.h"

#include "Utilities/KernelUtilities.h"
#include "Calculation/Calculation.h"
#include "Parameter/Parameter.h"

using namespace vf::basics::constant;
using namespace vf::lbm::dir;
using namespace vf::gpu;


// coarse to fine
template <bool hasTurbulentViscosity>
__global__ void scaleCoarseToFineCompressible_Device(
    real* distributionsCoarse,
    real* distributionsFine,
    uint* neighborXcoarse,
    uint* neighborYcoarse,
    uint* neighborZcoarse,
    uint* neighborXfine,
    uint* neighborYfine,
    uint* neighborZfine,
    unsigned long long numberOfLBnodesCoarse,
    unsigned long long numberOfLBnodesFine,
    bool isEvenTimestep,
    uint* indicesCoarseMMM,
    uint* indicesFineMMM,
    uint numberOfInterfaceNodes,
    real omegaCoarse,
    real omegaFine,
    real* turbulentViscosityCoarse,
    real* turbulentViscosityFine,
    ICellNeigh offsetCF);


__global__ void scaleCoarseToFineAdvectionDiffusion_Device(
    real* DC,
    real* DF,
    real* DD27C,
    real* DD27F,
    uint* neighborCX,
    uint* neighborCY,
    uint* neighborCZ,
    uint* neighborFX,
    uint* neighborFY,
    uint* neighborFZ,
    unsigned long long numberOfLBnodesC,
    unsigned long long numberOfLBnodesF,
    bool isEvenTimestep,
    uint* posCSWB,
    uint* posFSWB,
    uint kCF,
    real nu,
    real diffusivity_fine,
    ICellNeigh neighborCoarseToFine);


// fine to coarse
template <bool hasTurbulentViscosity>
__global__ void scaleFineToCoarseCompressible_Device(
    real* distributionsCoarse,
    real* distributionsFine,
    uint* neighborXcoarse,
    uint* neighborYcoarse,
    uint* neighborZcoarse,
    uint* neighborXfine,
    uint* neighborYfine,
    uint* neighborZfine,
    unsigned long long numberOfLBnodesCoarse,
    unsigned long long numberOfLBnodesFine,
    bool isEvenTimestep,
    uint* indicesCoarse000,
    uint* indicesFineMMM,
    uint numberOfInterfaceNodes,
    real omegaCoarse,
    real omegaFine,
    real* turbulentViscosityCoarse,
    real* turbulentViscosityFine,
    ICellNeigh offsetFC);


__global__ void scaleFineToCoarseAdvectionDiffusion_Device(
    real* DC,
    real* DF,
    real* DD27C,
    real* DD27F,
    uint* neighborCX,
    uint* neighborCY,
    uint* neighborCZ,
    uint* neighborFX,
    uint* neighborFY,
    uint* neighborFZ,
    unsigned long long numberOfLBnodesC,
    unsigned long long numberOfLBnodesF,
    bool isEvenTimestep,
    uint* posC,
    uint* posFSWB,
    uint kFC,
    real nu,
    real diffusivity_coarse,
    ICellNeigh neighborFineToCoarse);

//////////////////////////////////////////////////////////////////////////

template <bool hasTurbulentViscosity>
void scaleCoarseToFineCompressible(
    LBMSimulationParameter* parameterDeviceC,
    LBMSimulationParameter* parameterDeviceF,
    ICells* coarseToFine,
    ICellNeigh& neighborCoarseToFine,
    ihipStream_t* stream)
{
    dim3 grid = vf::cuda::getCudaGrid(parameterDeviceC->numberofthreads, coarseToFine->numberOfCells);
    dim3 threads(parameterDeviceC->numberofthreads, 1, 1);

    scaleCoarseToFineCompressible_Device<hasTurbulentViscosity><<<grid, threads, 0, stream>>>(
        parameterDeviceC->distributions.f[0],
        parameterDeviceF->distributions.f[0],
        parameterDeviceC->neighborX,
        parameterDeviceC->neighborY,
        parameterDeviceC->neighborZ,
        parameterDeviceF->neighborX,
        parameterDeviceF->neighborY,
        parameterDeviceF->neighborZ,
        parameterDeviceC->numberOfNodes,
        parameterDeviceF->numberOfNodes,
        parameterDeviceC->isEvenTimestep,
        coarseToFine->coarseCellIndices,
        coarseToFine->fineCellIndices,
        coarseToFine->numberOfCells,
        parameterDeviceC->omega,
        parameterDeviceF->omega,
        parameterDeviceC->turbViscosity,
        parameterDeviceF->turbViscosity,
        neighborCoarseToFine);

    getLastCudaError("scaleCoarseToFineCompressible_Device execution failed");
}
template void scaleCoarseToFineCompressible<true>(
    LBMSimulationParameter* parameterDeviceC,
    LBMSimulationParameter* parameterDeviceF,
    ICells* coarseToFine,
    ICellNeigh& neighborCoarseToFine,
    ihipStream_t* stream);
template void scaleCoarseToFineCompressible<false>(
    LBMSimulationParameter* parameterDeviceC,
    LBMSimulationParameter* parameterDeviceF,
    ICells* coarseToFine,
    ICellNeigh& neighborCoarseToFine,
    ihipStream_t* stream);

void scaleCoarseToFineAdvectionDiffusion(
    real* DC,
    real* DF,
    real* DD27C,
    real* DD27F,
    uint* neighborCX,
    uint* neighborCY,
    uint* neighborCZ,
    uint* neighborFX,
    uint* neighborFY,
    uint* neighborFZ,
    unsigned long long numberOfLBnodesC,
    unsigned long long numberOfLBnodesF,
    bool isEvenTimestep,
    uint* posCSWB,
    uint* posFSWB,
    uint kCF,
    real nu,
    real diffusivity_fine,
    uint numberOfThreads,
    ICellNeigh neighborCoarseToFine)
{
    vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kCF);

    scaleCoarseToFineAdvectionDiffusion_Device<<<grid.grid, grid.threads>>>(
        DC,
        DF,
        DD27C,
        DD27F,
        neighborCX,
        neighborCY,
        neighborCZ,
        neighborFX,
        neighborFY,
        neighborFZ,
        numberOfLBnodesC,
        numberOfLBnodesF,
        isEvenTimestep,
        posCSWB,
        posFSWB,
        kCF,
        nu,
        diffusivity_fine,
        neighborCoarseToFine);
    getLastCudaError("scaleCoarseToFineAdvectionDiffusion_Device execution failed");
}


template <bool hasTurbulentViscosity>
void scaleFineToCoarseCompressible(
    LBMSimulationParameter* parameterDeviceC,
    LBMSimulationParameter* parameterDeviceF,
    ICells* fineToCoarse,
    ICellNeigh& neighborFineToCoarse,
    ihipStream_t* stream)
{
    dim3 grid = vf::cuda::getCudaGrid(parameterDeviceC->numberofthreads, fineToCoarse->numberOfCells);
    dim3 threads(parameterDeviceC->numberofthreads, 1, 1);

    scaleFineToCoarseCompressible_Device<hasTurbulentViscosity><<<grid, threads, 0, stream>>>(
        parameterDeviceC->distributions.f[0],
        parameterDeviceF->distributions.f[0],
        parameterDeviceC->neighborX,
        parameterDeviceC->neighborY,
        parameterDeviceC->neighborZ,
        parameterDeviceF->neighborX,
        parameterDeviceF->neighborY,
        parameterDeviceF->neighborZ,
        parameterDeviceC->numberOfNodes,
        parameterDeviceF->numberOfNodes,
        parameterDeviceC->isEvenTimestep,
        fineToCoarse->coarseCellIndices,
        fineToCoarse->fineCellIndices,
        fineToCoarse->numberOfCells,
        parameterDeviceC->omega,
        parameterDeviceF->omega,
        parameterDeviceC->turbViscosity,
        parameterDeviceF->turbViscosity,
        neighborFineToCoarse);

    getLastCudaError("scaleFineToCoarseCompressible_Device execution failed");
}
template void scaleFineToCoarseCompressible<true>(
    LBMSimulationParameter* parameterDeviceC,
    LBMSimulationParameter* parameterDeviceF,
    ICells* fineToCoarse,
    ICellNeigh& neighborFineToCoarse,
    ihipStream_t* stream);
template void scaleFineToCoarseCompressible<false>(
    LBMSimulationParameter* parameterDeviceC,
    LBMSimulationParameter* parameterDeviceF,
    ICells* fineToCoarse,
    ICellNeigh& neighborFineToCoarse,
    ihipStream_t* stream);


void scaleFineToCoarseAdvectionDiffusion(
    real* DC,
    real* DF,
    real* DD27C,
    real* DD27F,
    uint* neighborCX,
    uint* neighborCY,
    uint* neighborCZ,
    uint* neighborFX,
    uint* neighborFY,
    uint* neighborFZ,
    unsigned long long numberOfLBnodesC,
    unsigned long long numberOfLBnodesF,
    bool isEvenTimestep,
    uint* posC,
    uint* posFSWB,
    uint kFC,
    real nu,
    real diffusivity_coarse,
    uint numberOfThreads,
    ICellNeigh neighborFineToCoarse)
{
    vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, kFC);

    scaleFineToCoarseAdvectionDiffusion_Device<<<grid.grid, grid.threads>>>(
        DC,
        DF,
        DD27C,
        DD27F,
        neighborCX,
        neighborCY,
        neighborCZ,
        neighborFX,
        neighborFY,
        neighborFZ,
        numberOfLBnodesC,
        numberOfLBnodesF,
        isEvenTimestep,
        posC,
        posFSWB,
        kFC,
        nu,
        diffusivity_coarse,
        neighborFineToCoarse);
    getLastCudaError("scaleFineToCoarseAdvectionDiffusion_Device execution failed");
}
