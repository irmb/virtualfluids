#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \author Martin Schoenherr
//=======================================================================================
#include "Calculation/Calculation.h" 
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;

__global__ void scaleCoarseToFineAdvectionDiffusion_Device(
    real* DC, 
    real* DF,
    real* DD27C, 
    real* DD27F, 
    uint* neighborCX,
    uint* neighborCY,
    uint* neighborCZ,
    uint* neighborFX,
    uint* neighborFY,
    uint* neighborFZ,
    unsigned long long numberOfLBnodesCoarse, 
    unsigned long long numberOfLBnodesFine, 
    bool isEvenTimestep,
    uint* posCSWB, 
    uint* posFSWB, 
    uint kCF, 
    real nu,
    real diffusivity_fine,
    ICellNeigh offCF)
{
   real *feF, *fwF, *fnF, *fsF, *ftF, *fbF, *fneF, *fswF, *fseF, *fnwF, *fteF, *fbwF, *fbeF, *ftwF, *ftnF, *fbsF, *fbnF, *ftsF, /**fzeroF,*/ *ftneF, *ftswF, *ftseF, *ftnwF, *fbneF, *fbswF, *fbseF, *fbnwF;

   feF    = &DF[dP00 * numberOfLBnodesFine];
   fwF    = &DF[dM00 * numberOfLBnodesFine];
   fnF    = &DF[d0P0 * numberOfLBnodesFine];
   fsF    = &DF[d0M0 * numberOfLBnodesFine];
   ftF    = &DF[d00P * numberOfLBnodesFine];
   fbF    = &DF[d00M * numberOfLBnodesFine];
   fneF   = &DF[dPP0 * numberOfLBnodesFine];
   fswF   = &DF[dMM0 * numberOfLBnodesFine];
   fseF   = &DF[dPM0 * numberOfLBnodesFine];
   fnwF   = &DF[dMP0 * numberOfLBnodesFine];
   fteF   = &DF[dP0P * numberOfLBnodesFine];
   fbwF   = &DF[dM0M * numberOfLBnodesFine];
   fbeF   = &DF[dP0M * numberOfLBnodesFine];
   ftwF   = &DF[dM0P * numberOfLBnodesFine];
   ftnF   = &DF[d0PP * numberOfLBnodesFine];
   fbsF   = &DF[d0MM * numberOfLBnodesFine];
   fbnF   = &DF[d0PM * numberOfLBnodesFine];
   ftsF   = &DF[d0MP * numberOfLBnodesFine];
   ftneF  = &DF[dPPP * numberOfLBnodesFine];
   ftswF  = &DF[dMMP * numberOfLBnodesFine];
   ftseF  = &DF[dPMP * numberOfLBnodesFine];
   ftnwF  = &DF[dMPP * numberOfLBnodesFine];
   fbneF  = &DF[dPPM * numberOfLBnodesFine];
   fbswF  = &DF[dMMM * numberOfLBnodesFine];
   fbseF  = &DF[dPMM * numberOfLBnodesFine];
   fbnwF  = &DF[dMPM * numberOfLBnodesFine];

   real *feC, *fwC, *fnC, *fsC, *ftC, *fbC, *fneC, *fswC, *fseC, *fnwC, *fteC, *fbwC, *fbeC, *ftwC, *ftnC, *fbsC, *fbnC, *ftsC, //*fzeroC,
      *ftneC, *ftswC, *ftseC, *ftnwC, *fbneC, *fbswC, *fbseC, *fbnwC;

   if (isEvenTimestep==true)
   {
      feC    = &DC[dP00 * numberOfLBnodesCoarse];
      fwC    = &DC[dM00 * numberOfLBnodesCoarse];
      fnC    = &DC[d0P0 * numberOfLBnodesCoarse];
      fsC    = &DC[d0M0 * numberOfLBnodesCoarse];
      ftC    = &DC[d00P * numberOfLBnodesCoarse];
      fbC    = &DC[d00M * numberOfLBnodesCoarse];
      fneC   = &DC[dPP0 * numberOfLBnodesCoarse];
      fswC   = &DC[dMM0 * numberOfLBnodesCoarse];
      fseC   = &DC[dPM0 * numberOfLBnodesCoarse];
      fnwC   = &DC[dMP0 * numberOfLBnodesCoarse];
      fteC   = &DC[dP0P * numberOfLBnodesCoarse];
      fbwC   = &DC[dM0M * numberOfLBnodesCoarse];
      fbeC   = &DC[dP0M * numberOfLBnodesCoarse];
      ftwC   = &DC[dM0P * numberOfLBnodesCoarse];
      ftnC   = &DC[d0PP * numberOfLBnodesCoarse];
      fbsC   = &DC[d0MM * numberOfLBnodesCoarse];
      fbnC   = &DC[d0PM * numberOfLBnodesCoarse];
      ftsC   = &DC[d0MP * numberOfLBnodesCoarse];
      ftneC  = &DC[dPPP * numberOfLBnodesCoarse];
      ftswC  = &DC[dMMP * numberOfLBnodesCoarse];
      ftseC  = &DC[dPMP * numberOfLBnodesCoarse];
      ftnwC  = &DC[dMPP * numberOfLBnodesCoarse];
      fbneC  = &DC[dPPM * numberOfLBnodesCoarse];
      fbswC  = &DC[dMMM * numberOfLBnodesCoarse];
      fbseC  = &DC[dPMM * numberOfLBnodesCoarse];
      fbnwC  = &DC[dMPM * numberOfLBnodesCoarse];
   } 
   else
   {
      fwC    = &DC[dP00 * numberOfLBnodesCoarse];
      feC    = &DC[dM00 * numberOfLBnodesCoarse];
      fsC    = &DC[d0P0 * numberOfLBnodesCoarse];
      fnC    = &DC[d0M0 * numberOfLBnodesCoarse];
      fbC    = &DC[d00P * numberOfLBnodesCoarse];
      ftC    = &DC[d00M * numberOfLBnodesCoarse];
      fswC   = &DC[dPP0 * numberOfLBnodesCoarse];
      fneC   = &DC[dMM0 * numberOfLBnodesCoarse];
      fnwC   = &DC[dPM0 * numberOfLBnodesCoarse];
      fseC   = &DC[dMP0 * numberOfLBnodesCoarse];
      fbwC   = &DC[dP0P * numberOfLBnodesCoarse];
      fteC   = &DC[dM0M * numberOfLBnodesCoarse];
      ftwC   = &DC[dP0M * numberOfLBnodesCoarse];
      fbeC   = &DC[dM0P * numberOfLBnodesCoarse];
      fbsC   = &DC[d0PP * numberOfLBnodesCoarse];
      ftnC   = &DC[d0MM * numberOfLBnodesCoarse];
      ftsC   = &DC[d0PM * numberOfLBnodesCoarse];
      fbnC   = &DC[d0MP * numberOfLBnodesCoarse];
      fbswC  = &DC[dPPP * numberOfLBnodesCoarse];
      fbneC  = &DC[dMMP * numberOfLBnodesCoarse];
      fbnwC  = &DC[dPMP * numberOfLBnodesCoarse];
      fbseC  = &DC[dMPP * numberOfLBnodesCoarse];
      ftswC  = &DC[dPPM * numberOfLBnodesCoarse];
      ftneC  = &DC[dMMM * numberOfLBnodesCoarse];
      ftnwC  = &DC[dPMM * numberOfLBnodesCoarse];
      ftseC  = &DC[dMPM * numberOfLBnodesCoarse];
   }


   Distributions27 D27F;
   D27F.f[dP00] = &DD27F[dP00 * numberOfLBnodesFine];
   D27F.f[dM00] = &DD27F[dM00 * numberOfLBnodesFine];
   D27F.f[d0P0] = &DD27F[d0P0 * numberOfLBnodesFine];
   D27F.f[d0M0] = &DD27F[d0M0 * numberOfLBnodesFine];
   D27F.f[d00P] = &DD27F[d00P * numberOfLBnodesFine];
   D27F.f[d00M] = &DD27F[d00M * numberOfLBnodesFine];
   D27F.f[dPP0] = &DD27F[dPP0 * numberOfLBnodesFine];
   D27F.f[dMM0] = &DD27F[dMM0 * numberOfLBnodesFine];
   D27F.f[dPM0] = &DD27F[dPM0 * numberOfLBnodesFine];
   D27F.f[dMP0] = &DD27F[dMP0 * numberOfLBnodesFine];
   D27F.f[dP0P] = &DD27F[dP0P * numberOfLBnodesFine];
   D27F.f[dM0M] = &DD27F[dM0M * numberOfLBnodesFine];
   D27F.f[dP0M] = &DD27F[dP0M * numberOfLBnodesFine];
   D27F.f[dM0P] = &DD27F[dM0P * numberOfLBnodesFine];
   D27F.f[d0PP] = &DD27F[d0PP * numberOfLBnodesFine];
   D27F.f[d0MM] = &DD27F[d0MM * numberOfLBnodesFine];
   D27F.f[d0PM] = &DD27F[d0PM * numberOfLBnodesFine];
   D27F.f[d0MP] = &DD27F[d0MP * numberOfLBnodesFine];
   D27F.f[d000] = &DD27F[d000 * numberOfLBnodesFine];
   D27F.f[dPPP] = &DD27F[dPPP * numberOfLBnodesFine];
   D27F.f[dMMP] = &DD27F[dMMP * numberOfLBnodesFine];
   D27F.f[dPMP] = &DD27F[dPMP * numberOfLBnodesFine];
   D27F.f[dMPP] = &DD27F[dMPP * numberOfLBnodesFine];
   D27F.f[dPPM] = &DD27F[dPPM * numberOfLBnodesFine];
   D27F.f[dMMM] = &DD27F[dMMM * numberOfLBnodesFine];
   D27F.f[dPMM] = &DD27F[dPMM * numberOfLBnodesFine];
   D27F.f[dMPM] = &DD27F[dMPM * numberOfLBnodesFine];

   Distributions27 D27C;
   if (isEvenTimestep==true)
   {
      D27C.f[dP00] = &DD27C[dP00 * numberOfLBnodesCoarse];
      D27C.f[dM00] = &DD27C[dM00 * numberOfLBnodesCoarse];
      D27C.f[d0P0] = &DD27C[d0P0 * numberOfLBnodesCoarse];
      D27C.f[d0M0] = &DD27C[d0M0 * numberOfLBnodesCoarse];
      D27C.f[d00P] = &DD27C[d00P * numberOfLBnodesCoarse];
      D27C.f[d00M] = &DD27C[d00M * numberOfLBnodesCoarse];
      D27C.f[dPP0] = &DD27C[dPP0 * numberOfLBnodesCoarse];
      D27C.f[dMM0] = &DD27C[dMM0 * numberOfLBnodesCoarse];
      D27C.f[dPM0] = &DD27C[dPM0 * numberOfLBnodesCoarse];
      D27C.f[dMP0] = &DD27C[dMP0 * numberOfLBnodesCoarse];
      D27C.f[dP0P] = &DD27C[dP0P * numberOfLBnodesCoarse];
      D27C.f[dM0M] = &DD27C[dM0M * numberOfLBnodesCoarse];
      D27C.f[dP0M] = &DD27C[dP0M * numberOfLBnodesCoarse];
      D27C.f[dM0P] = &DD27C[dM0P * numberOfLBnodesCoarse];
      D27C.f[d0PP] = &DD27C[d0PP * numberOfLBnodesCoarse];
      D27C.f[d0MM] = &DD27C[d0MM * numberOfLBnodesCoarse];
      D27C.f[d0PM] = &DD27C[d0PM * numberOfLBnodesCoarse];
      D27C.f[d0MP] = &DD27C[d0MP * numberOfLBnodesCoarse];
      D27C.f[d000] = &DD27C[d000 * numberOfLBnodesCoarse];
      D27C.f[dPPP] = &DD27C[dPPP * numberOfLBnodesCoarse];
      D27C.f[dMMP] = &DD27C[dMMP * numberOfLBnodesCoarse];
      D27C.f[dPMP] = &DD27C[dPMP * numberOfLBnodesCoarse];
      D27C.f[dMPP] = &DD27C[dMPP * numberOfLBnodesCoarse];
      D27C.f[dPPM] = &DD27C[dPPM * numberOfLBnodesCoarse];
      D27C.f[dMMM] = &DD27C[dMMM * numberOfLBnodesCoarse];
      D27C.f[dPMM] = &DD27C[dPMM * numberOfLBnodesCoarse];
      D27C.f[dMPM] = &DD27C[dMPM * numberOfLBnodesCoarse];
   }
   else
   {
      D27C.f[dM00] = &DD27C[dP00 * numberOfLBnodesCoarse];
      D27C.f[dP00] = &DD27C[dM00 * numberOfLBnodesCoarse];
      D27C.f[d0M0] = &DD27C[d0P0 * numberOfLBnodesCoarse];
      D27C.f[d0P0] = &DD27C[d0M0 * numberOfLBnodesCoarse];
      D27C.f[d00M] = &DD27C[d00P * numberOfLBnodesCoarse];
      D27C.f[d00P] = &DD27C[d00M * numberOfLBnodesCoarse];
      D27C.f[dMM0] = &DD27C[dPP0 * numberOfLBnodesCoarse];
      D27C.f[dPP0] = &DD27C[dMM0 * numberOfLBnodesCoarse];
      D27C.f[dMP0] = &DD27C[dPM0 * numberOfLBnodesCoarse];
      D27C.f[dPM0] = &DD27C[dMP0 * numberOfLBnodesCoarse];
      D27C.f[dM0M] = &DD27C[dP0P * numberOfLBnodesCoarse];
      D27C.f[dP0P] = &DD27C[dM0M * numberOfLBnodesCoarse];
      D27C.f[dM0P] = &DD27C[dP0M * numberOfLBnodesCoarse];
      D27C.f[dP0M] = &DD27C[dM0P * numberOfLBnodesCoarse];
      D27C.f[d0MM] = &DD27C[d0PP * numberOfLBnodesCoarse];
      D27C.f[d0PP] = &DD27C[d0MM * numberOfLBnodesCoarse];
      D27C.f[d0MP] = &DD27C[d0PM * numberOfLBnodesCoarse];
      D27C.f[d0PM] = &DD27C[d0MP * numberOfLBnodesCoarse];
      D27C.f[d000] = &DD27C[d000 * numberOfLBnodesCoarse];
      D27C.f[dMMM] = &DD27C[dPPP * numberOfLBnodesCoarse];
      D27C.f[dPPM] = &DD27C[dMMP * numberOfLBnodesCoarse];
      D27C.f[dMPM] = &DD27C[dPMP * numberOfLBnodesCoarse];
      D27C.f[dPMM] = &DD27C[dMPP * numberOfLBnodesCoarse];
      D27C.f[dMMP] = &DD27C[dPPM * numberOfLBnodesCoarse];
      D27C.f[dPPP] = &DD27C[dMMM * numberOfLBnodesCoarse];
      D27C.f[dMPP] = &DD27C[dPMM * numberOfLBnodesCoarse];
      D27C.f[dPMP] = &DD27C[dMPM * numberOfLBnodesCoarse];
   }

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  ix = threadIdx.x;
   const unsigned  iy = blockIdx.x; 
   const unsigned  iz = blockIdx.y; 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*iz + iy) + ix;
   //////////////////////////////////////////////////////////////////////////

   ////////////////////////////////////////////////////////////////////////////////
   real vx1,vx2,vx3, cu_sq;
   real f_E,f_W,f_N,f_S,f_T,f_B,f_NE,f_SW,f_SE,f_NW,f_TE,f_BW,f_BE,f_TW,f_TN,f_BS,f_BN,f_TS,f_TNE,f_TSW,f_TSE,f_TNW,f_BNE,f_BSW,f_BSE,f_BNW;

   real f27E,f27W,f27N,f27S,f27T,f27B,f27NE,f27SW,f27SE,f27NW,f27TE,f27BW,f27BE,f27TW,f27TN,f27BS,f27BN,f27TS,f27ZERO,f27TNE,f27TSW,f27TSE,f27TNW,f27BNE,f27BSW,f27BSE,f27BNW;
   real Mx,My,Mz; 
   real Conc_C_SWB, Conc_C_SWT, Conc_C_SET, Conc_C_SEB, Conc_C_NWB, Conc_C_NWT, Conc_C_NET, Conc_C_NEB;

   real omegaD_C     = c2o1 / (c6o1 * diffusivity_fine/c2o1 + c1o1);
   real omegaD_F     = c2o1 / (c6o1 * diffusivity_fine + c1o1);

   real x,       y,       z;
   real xoff,    yoff,    zoff;
   real xoff_sq, yoff_sq, zoff_sq;

   if(k<kCF)
   {
      //////////////////////////////////////////////////////////////////////////
      xoff    = offCF.x[k];
      yoff    = offCF.y[k];
      zoff    = offCF.z[k];
      xoff_sq = xoff * xoff;
      yoff_sq = yoff * yoff;
      zoff_sq = zoff * zoff;
      //////////////////////////////////////////////////////////////////////////
      //SWB//
      //////////////////////////////////////////////////////////////////////////
      //index 0
      unsigned int k0zero= posCSWB[k];
      unsigned int k0w   = neighborCX[k0zero];
      unsigned int k0s   = neighborCY[k0zero];
      unsigned int k0b   = neighborCZ[k0zero];
      unsigned int k0sw  = neighborCY[k0w];
      unsigned int k0bw  = neighborCZ[k0w];
      unsigned int k0bs  = neighborCZ[k0s];
      unsigned int k0bsw = neighborCZ[k0sw];
      //////////////////////////////////////////////////////////////////////////
      //index 
      unsigned int kzero= k0zero;
      unsigned int kw   = k0w;   
      unsigned int ks   = k0s;   
      unsigned int kb   = k0b;   
      unsigned int ksw  = k0sw;  
      unsigned int kbw  = k0bw;  
      unsigned int kbs  = k0bs;  
      unsigned int kbsw = k0bsw; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feC[kzero];
      f_W    = fwC[kw];
      f_N    = fnC[kzero];
      f_S    = fsC[ks];
      f_T    = ftC[kzero];
      f_B    = fbC[kb];
      f_NE   = fneC[kzero];
      f_SW   = fswC[ksw];
      f_SE   = fseC[ks];
      f_NW   = fnwC[kw];
      f_TE   = fteC[kzero];
      f_BW   = fbwC[kbw];
      f_BE   = fbeC[kb];
      f_TW   = ftwC[kw];
      f_TN   = ftnC[kzero];
      f_BS   = fbsC[kbs];
      f_BN   = fbnC[kb];
      f_TS   = ftsC[ks];
      f_TNE  = ftneC[kzero];
      f_TSW  = ftswC[ksw];
      f_TSE  = ftseC[ks];
      f_TNW  = ftnwC[kw];
      f_BNE  = fbneC[kb];
      f_BSW  = fbswC[kbsw];
      f_BSE  = fbseC[kbs];
      f_BNW  = fbnwC[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27C.f[dP00])[kzero];//ke
      f27W    =  (D27C.f[dM00])[kw   ];
      f27N    =  (D27C.f[d0P0])[kzero];//kn
      f27S    =  (D27C.f[d0M0])[ks   ];
      f27T    =  (D27C.f[d00P])[kzero];//kt
      f27B    =  (D27C.f[d00M])[kb   ];
      f27NE   =  (D27C.f[dPP0])[kzero];//kne
      f27SW   =  (D27C.f[dMM0])[ksw  ];
      f27SE   =  (D27C.f[dPM0])[ks   ];//kse
      f27NW   =  (D27C.f[dMP0])[kw   ];//knw
      f27TE   =  (D27C.f[dP0P])[kzero];//kte
      f27BW   =  (D27C.f[dM0M])[kbw  ];
      f27BE   =  (D27C.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27C.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27C.f[d0PP])[kzero];//ktn
      f27BS   =  (D27C.f[d0MM])[kbs  ];
      f27BN   =  (D27C.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27C.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27C.f[d000])[kzero];//kzero
      f27TNE   = (D27C.f[dPPP])[kzero];//ktne
      f27TSW   = (D27C.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27C.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27C.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27C.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27C.f[dMMM])[kbsw ];
      f27BSE   = (D27C.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27C.f[dMPM])[kbw  ];//kbnw

      Conc_C_SWB = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW; 

      vx1  =f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  =f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  =f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_CSWB = (Conc_C_SWB * vx1 - Mx) * (c3o1*omegaD_C);
      real Diff_Conc_Y_CSWB = (Conc_C_SWB * vx2 - My) * (c3o1*omegaD_C);
      real Diff_Conc_Z_CSWB = (Conc_C_SWB * vx3 - Mz) * (c3o1*omegaD_C);




      //////////////////////////////////////////////////////////////////////////
      //SWT//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kb;
      kw   = kbw;   
      ks   = kbs;   
      kb   = neighborCZ[kb];   
      ksw  = kbsw;  
      kbw  = neighborCZ[kbw];  
      kbs  = neighborCZ[kbs];  
      kbsw = neighborCZ[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feC[kzero];
      f_W    = fwC[kw];
      f_N    = fnC[kzero];
      f_S    = fsC[ks];
      f_T    = ftC[kzero];
      f_B    = fbC[kb];
      f_NE   = fneC[kzero];
      f_SW   = fswC[ksw];
      f_SE   = fseC[ks];
      f_NW   = fnwC[kw];
      f_TE   = fteC[kzero];
      f_BW   = fbwC[kbw];
      f_BE   = fbeC[kb];
      f_TW   = ftwC[kw];
      f_TN   = ftnC[kzero];
      f_BS   = fbsC[kbs];
      f_BN   = fbnC[kb];
      f_TS   = ftsC[ks];
      f_TNE  = ftneC[kzero];
      f_TSW  = ftswC[ksw];
      f_TSE  = ftseC[ks];
      f_TNW  = ftnwC[kw];
      f_BNE  = fbneC[kb];
      f_BSW  = fbswC[kbsw];
      f_BSE  = fbseC[kbs];
      f_BNW  = fbnwC[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27C.f[dP00])[kzero];//ke
      f27W    =  (D27C.f[dM00])[kw   ];
      f27N    =  (D27C.f[d0P0])[kzero];//kn
      f27S    =  (D27C.f[d0M0])[ks   ];
      f27T    =  (D27C.f[d00P])[kzero];//kt
      f27B    =  (D27C.f[d00M])[kb   ];
      f27NE   =  (D27C.f[dPP0])[kzero];//kne
      f27SW   =  (D27C.f[dMM0])[ksw  ];
      f27SE   =  (D27C.f[dPM0])[ks   ];//kse
      f27NW   =  (D27C.f[dMP0])[kw   ];//knw
      f27TE   =  (D27C.f[dP0P])[kzero];//kte
      f27BW   =  (D27C.f[dM0M])[kbw  ];
      f27BE   =  (D27C.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27C.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27C.f[d0PP])[kzero];//ktn
      f27BS   =  (D27C.f[d0MM])[kbs  ];
      f27BN   =  (D27C.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27C.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27C.f[d000])[kzero];//kzero
      f27TNE   = (D27C.f[dPPP])[kzero];//ktne
      f27TSW   = (D27C.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27C.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27C.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27C.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27C.f[dMMM])[kbsw ];
      f27BSE   = (D27C.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27C.f[dMPM])[kbw  ];//kbnw

      Conc_C_SWT = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW; 

      vx1  =f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  =f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  =f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_CSWT = (Conc_C_SWT * vx1 - Mx) * (c3o1*omegaD_C);
      real Diff_Conc_Y_CSWT = (Conc_C_SWT * vx2 - My) * (c3o1*omegaD_C);
      real Diff_Conc_Z_CSWT = (Conc_C_SWT * vx3 - Mz) * (c3o1*omegaD_C);




      //////////////////////////////////////////////////////////////////////////
      //SET//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kw;
      kw   = neighborCX[kw];   
      ks   = ksw;   
      kb   = kbw;   
      ksw  = neighborCX[ksw];  
      kbw  = neighborCX[kbw];  
      kbs  = kbsw;  
      kbsw = neighborCX[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feC[kzero];
      f_W    = fwC[kw];
      f_N    = fnC[kzero];
      f_S    = fsC[ks];
      f_T    = ftC[kzero];
      f_B    = fbC[kb];
      f_NE   = fneC[kzero];
      f_SW   = fswC[ksw];
      f_SE   = fseC[ks];
      f_NW   = fnwC[kw];
      f_TE   = fteC[kzero];
      f_BW   = fbwC[kbw];
      f_BE   = fbeC[kb];
      f_TW   = ftwC[kw];
      f_TN   = ftnC[kzero];
      f_BS   = fbsC[kbs];
      f_BN   = fbnC[kb];
      f_TS   = ftsC[ks];
      f_TNE  = ftneC[kzero];
      f_TSW  = ftswC[ksw];
      f_TSE  = ftseC[ks];
      f_TNW  = ftnwC[kw];
      f_BNE  = fbneC[kb];
      f_BSW  = fbswC[kbsw];
      f_BSE  = fbseC[kbs];
      f_BNW  = fbnwC[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27C.f[dP00])[kzero];//ke
      f27W    =  (D27C.f[dM00])[kw   ];
      f27N    =  (D27C.f[d0P0])[kzero];//kn
      f27S    =  (D27C.f[d0M0])[ks   ];
      f27T    =  (D27C.f[d00P])[kzero];//kt
      f27B    =  (D27C.f[d00M])[kb   ];
      f27NE   =  (D27C.f[dPP0])[kzero];//kne
      f27SW   =  (D27C.f[dMM0])[ksw  ];
      f27SE   =  (D27C.f[dPM0])[ks   ];//kse
      f27NW   =  (D27C.f[dMP0])[kw   ];//knw
      f27TE   =  (D27C.f[dP0P])[kzero];//kte
      f27BW   =  (D27C.f[dM0M])[kbw  ];
      f27BE   =  (D27C.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27C.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27C.f[d0PP])[kzero];//ktn
      f27BS   =  (D27C.f[d0MM])[kbs  ];
      f27BN   =  (D27C.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27C.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27C.f[d000])[kzero];//kzero
      f27TNE   = (D27C.f[dPPP])[kzero];//ktne
      f27TSW   = (D27C.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27C.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27C.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27C.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27C.f[dMMM])[kbsw ];
      f27BSE   = (D27C.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27C.f[dMPM])[kbw  ];//kbnw

      Conc_C_SET = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW; 

      vx1  =f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  =f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  =f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_CSET = (Conc_C_SET * vx1 - Mx) * (c3o1*omegaD_C);
      real Diff_Conc_Y_CSET = (Conc_C_SET * vx2 - My) * (c3o1*omegaD_C);
      real Diff_Conc_Z_CSET = (Conc_C_SET * vx3 - Mz) * (c3o1*omegaD_C);




      //////////////////////////////////////////////////////////////////////////
      //SEB//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kb   = kzero;   
      kbw  = kw;  
      kbs  = ks;  
      kbsw = ksw; 
      kzero= k0w;
      kw   = neighborCX[k0w];   
      ks   = k0sw;   
      ksw  = neighborCX[k0sw];  
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feC[kzero];
      f_W    = fwC[kw];
      f_N    = fnC[kzero];
      f_S    = fsC[ks];
      f_T    = ftC[kzero];
      f_B    = fbC[kb];
      f_NE   = fneC[kzero];
      f_SW   = fswC[ksw];
      f_SE   = fseC[ks];
      f_NW   = fnwC[kw];
      f_TE   = fteC[kzero];
      f_BW   = fbwC[kbw];
      f_BE   = fbeC[kb];
      f_TW   = ftwC[kw];
      f_TN   = ftnC[kzero];
      f_BS   = fbsC[kbs];
      f_BN   = fbnC[kb];
      f_TS   = ftsC[ks];
      f_TNE  = ftneC[kzero];
      f_TSW  = ftswC[ksw];
      f_TSE  = ftseC[ks];
      f_TNW  = ftnwC[kw];
      f_BNE  = fbneC[kb];
      f_BSW  = fbswC[kbsw];
      f_BSE  = fbseC[kbs];
      f_BNW  = fbnwC[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27C.f[dP00])[kzero];//ke
      f27W    =  (D27C.f[dM00])[kw   ];
      f27N    =  (D27C.f[d0P0])[kzero];//kn
      f27S    =  (D27C.f[d0M0])[ks   ];
      f27T    =  (D27C.f[d00P])[kzero];//kt
      f27B    =  (D27C.f[d00M])[kb   ];
      f27NE   =  (D27C.f[dPP0])[kzero];//kne
      f27SW   =  (D27C.f[dMM0])[ksw  ];
      f27SE   =  (D27C.f[dPM0])[ks   ];//kse
      f27NW   =  (D27C.f[dMP0])[kw   ];//knw
      f27TE   =  (D27C.f[dP0P])[kzero];//kte
      f27BW   =  (D27C.f[dM0M])[kbw  ];
      f27BE   =  (D27C.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27C.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27C.f[d0PP])[kzero];//ktn
      f27BS   =  (D27C.f[d0MM])[kbs  ];
      f27BN   =  (D27C.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27C.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27C.f[d000])[kzero];//kzero
      f27TNE   = (D27C.f[dPPP])[kzero];//ktne
      f27TSW   = (D27C.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27C.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27C.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27C.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27C.f[dMMM])[kbsw ];
      f27BSE   = (D27C.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27C.f[dMPM])[kbw  ];//kbnw

      Conc_C_SEB = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW; 

      vx1  =f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  =f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  =f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_CSEB = (Conc_C_SEB * vx1 - Mx) * (c3o1*omegaD_C);
      real Diff_Conc_Y_CSEB = (Conc_C_SEB * vx2 - My) * (c3o1*omegaD_C);
      real Diff_Conc_Z_CSEB = (Conc_C_SEB * vx3 - Mz) * (c3o1*omegaD_C);




      //////////////////////////////////////////////////////////////////////////
      //NWB//
      //////////////////////////////////////////////////////////////////////////
      //index 0
      k0zero= k0s;
      k0w   = k0sw;
      k0s   = neighborCY[k0s];
      k0b   = k0bs;
      k0sw  = neighborCY[k0sw];
      k0bw  = k0bsw;
      k0bs  = neighborCY[k0bs];
      k0bsw = neighborCY[k0bsw];
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= k0zero;
      kw   = k0w;   
      ks   = k0s;   
      kb   = k0b;   
      ksw  = k0sw;  
      kbw  = k0bw;  
      kbs  = k0bs;  
      kbsw = k0bsw; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feC[kzero];
      f_W    = fwC[kw];
      f_N    = fnC[kzero];
      f_S    = fsC[ks];
      f_T    = ftC[kzero];
      f_B    = fbC[kb];
      f_NE   = fneC[kzero];
      f_SW   = fswC[ksw];
      f_SE   = fseC[ks];
      f_NW   = fnwC[kw];
      f_TE   = fteC[kzero];
      f_BW   = fbwC[kbw];
      f_BE   = fbeC[kb];
      f_TW   = ftwC[kw];
      f_TN   = ftnC[kzero];
      f_BS   = fbsC[kbs];
      f_BN   = fbnC[kb];
      f_TS   = ftsC[ks];
      f_TNE  = ftneC[kzero];
      f_TSW  = ftswC[ksw];
      f_TSE  = ftseC[ks];
      f_TNW  = ftnwC[kw];
      f_BNE  = fbneC[kb];
      f_BSW  = fbswC[kbsw];
      f_BSE  = fbseC[kbs];
      f_BNW  = fbnwC[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27C.f[dP00])[kzero];//ke
      f27W    =  (D27C.f[dM00])[kw   ];
      f27N    =  (D27C.f[d0P0])[kzero];//kn
      f27S    =  (D27C.f[d0M0])[ks   ];
      f27T    =  (D27C.f[d00P])[kzero];//kt
      f27B    =  (D27C.f[d00M])[kb   ];
      f27NE   =  (D27C.f[dPP0])[kzero];//kne
      f27SW   =  (D27C.f[dMM0])[ksw  ];
      f27SE   =  (D27C.f[dPM0])[ks   ];//kse
      f27NW   =  (D27C.f[dMP0])[kw   ];//knw
      f27TE   =  (D27C.f[dP0P])[kzero];//kte
      f27BW   =  (D27C.f[dM0M])[kbw  ];
      f27BE   =  (D27C.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27C.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27C.f[d0PP])[kzero];//ktn
      f27BS   =  (D27C.f[d0MM])[kbs  ];
      f27BN   =  (D27C.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27C.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27C.f[d000])[kzero];//kzero
      f27TNE   = (D27C.f[dPPP])[kzero];//ktne
      f27TSW   = (D27C.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27C.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27C.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27C.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27C.f[dMMM])[kbsw ];
      f27BSE   = (D27C.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27C.f[dMPM])[kbw  ];//kbnw

      Conc_C_NWB = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW; 

      vx1  =f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  =f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  =f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_CNWB = (Conc_C_NWB * vx1 - Mx) * (c3o1*omegaD_C);
      real Diff_Conc_Y_CNWB = (Conc_C_NWB * vx2 - My) * (c3o1*omegaD_C);
      real Diff_Conc_Z_CNWB = (Conc_C_NWB * vx3 - Mz) * (c3o1*omegaD_C);




      //////////////////////////////////////////////////////////////////////////
      //NWT//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kb;
      kw   = kbw;   
      ks   = kbs;   
      kb   = neighborCZ[kb];   
      ksw  = kbsw;  
      kbw  = neighborCZ[kbw];  
      kbs  = neighborCZ[kbs];  
      kbsw = neighborCZ[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feC[kzero];
      f_W    = fwC[kw];
      f_N    = fnC[kzero];
      f_S    = fsC[ks];
      f_T    = ftC[kzero];
      f_B    = fbC[kb];
      f_NE   = fneC[kzero];
      f_SW   = fswC[ksw];
      f_SE   = fseC[ks];
      f_NW   = fnwC[kw];
      f_TE   = fteC[kzero];
      f_BW   = fbwC[kbw];
      f_BE   = fbeC[kb];
      f_TW   = ftwC[kw];
      f_TN   = ftnC[kzero];
      f_BS   = fbsC[kbs];
      f_BN   = fbnC[kb];
      f_TS   = ftsC[ks];
      f_TNE  = ftneC[kzero];
      f_TSW  = ftswC[ksw];
      f_TSE  = ftseC[ks];
      f_TNW  = ftnwC[kw];
      f_BNE  = fbneC[kb];
      f_BSW  = fbswC[kbsw];
      f_BSE  = fbseC[kbs];
      f_BNW  = fbnwC[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27C.f[dP00])[kzero];//ke
      f27W    =  (D27C.f[dM00])[kw   ];
      f27N    =  (D27C.f[d0P0])[kzero];//kn
      f27S    =  (D27C.f[d0M0])[ks   ];
      f27T    =  (D27C.f[d00P])[kzero];//kt
      f27B    =  (D27C.f[d00M])[kb   ];
      f27NE   =  (D27C.f[dPP0])[kzero];//kne
      f27SW   =  (D27C.f[dMM0])[ksw  ];
      f27SE   =  (D27C.f[dPM0])[ks   ];//kse
      f27NW   =  (D27C.f[dMP0])[kw   ];//knw
      f27TE   =  (D27C.f[dP0P])[kzero];//kte
      f27BW   =  (D27C.f[dM0M])[kbw  ];
      f27BE   =  (D27C.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27C.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27C.f[d0PP])[kzero];//ktn
      f27BS   =  (D27C.f[d0MM])[kbs  ];
      f27BN   =  (D27C.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27C.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27C.f[d000])[kzero];//kzero
      f27TNE   = (D27C.f[dPPP])[kzero];//ktne
      f27TSW   = (D27C.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27C.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27C.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27C.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27C.f[dMMM])[kbsw ];
      f27BSE   = (D27C.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27C.f[dMPM])[kbw  ];//kbnw

      Conc_C_NWT = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW; 

      vx1  =f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  =f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  =f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_CNWT = (Conc_C_NWT * vx1 - Mx) * (c3o1*omegaD_C);
      real Diff_Conc_Y_CNWT = (Conc_C_NWT * vx2 - My) * (c3o1*omegaD_C);
      real Diff_Conc_Z_CNWT = (Conc_C_NWT * vx3 - Mz) * (c3o1*omegaD_C);




      //////////////////////////////////////////////////////////////////////////
      //NET//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kw;
      kw   = neighborCX[kw];   
      ks   = ksw;   
      kb   = kbw;   
      ksw  = neighborCX[ksw];  
      kbw  = neighborCX[kbw];  
      kbs  = kbsw;  
      kbsw = neighborCX[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feC[kzero];
      f_W    = fwC[kw];
      f_N    = fnC[kzero];
      f_S    = fsC[ks];
      f_T    = ftC[kzero];
      f_B    = fbC[kb];
      f_NE   = fneC[kzero];
      f_SW   = fswC[ksw];
      f_SE   = fseC[ks];
      f_NW   = fnwC[kw];
      f_TE   = fteC[kzero];
      f_BW   = fbwC[kbw];
      f_BE   = fbeC[kb];
      f_TW   = ftwC[kw];
      f_TN   = ftnC[kzero];
      f_BS   = fbsC[kbs];
      f_BN   = fbnC[kb];
      f_TS   = ftsC[ks];
      f_TNE  = ftneC[kzero];
      f_TSW  = ftswC[ksw];
      f_TSE  = ftseC[ks];
      f_TNW  = ftnwC[kw];
      f_BNE  = fbneC[kb];
      f_BSW  = fbswC[kbsw];
      f_BSE  = fbseC[kbs];
      f_BNW  = fbnwC[kbw];
      //////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27C.f[dP00])[kzero];//ke
      f27W    =  (D27C.f[dM00])[kw   ];
      f27N    =  (D27C.f[d0P0])[kzero];//kn
      f27S    =  (D27C.f[d0M0])[ks   ];
      f27T    =  (D27C.f[d00P])[kzero];//kt
      f27B    =  (D27C.f[d00M])[kb   ];
      f27NE   =  (D27C.f[dPP0])[kzero];//kne
      f27SW   =  (D27C.f[dMM0])[ksw  ];
      f27SE   =  (D27C.f[dPM0])[ks   ];//kse
      f27NW   =  (D27C.f[dMP0])[kw   ];//knw
      f27TE   =  (D27C.f[dP0P])[kzero];//kte
      f27BW   =  (D27C.f[dM0M])[kbw  ];
      f27BE   =  (D27C.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27C.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27C.f[d0PP])[kzero];//ktn
      f27BS   =  (D27C.f[d0MM])[kbs  ];
      f27BN   =  (D27C.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27C.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27C.f[d000])[kzero];//kzero
      f27TNE   = (D27C.f[dPPP])[kzero];//ktne
      f27TSW   = (D27C.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27C.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27C.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27C.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27C.f[dMMM])[kbsw ];
      f27BSE   = (D27C.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27C.f[dMPM])[kbw  ];//kbnw

      Conc_C_NET = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW; 

      vx1  =f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  =f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  =f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_CNET = (Conc_C_NET * vx1 - Mx) * (c3o1*omegaD_C);
      real Diff_Conc_Y_CNET = (Conc_C_NET * vx2 - My) * (c3o1*omegaD_C);
      real Diff_Conc_Z_CNET = (Conc_C_NET * vx3 - Mz) * (c3o1*omegaD_C);




      //////////////////////////////////////////////////////////////////////////
      //NEB//
      //////////////////////////////////////////////////////////////////////////
      //index 
      kb   = kzero;   
      kbw  = kw;  
      kbs  = ks;  
      kbsw = ksw; 
      kzero= k0w;
      kw   = neighborCX[k0w];   
      ks   = k0sw;   
      ksw  = neighborCX[k0sw];  
      ////////////////////////////////////////////////////////////////////////////////
      f_E    = feC[kzero];
      f_W    = fwC[kw];
      f_N    = fnC[kzero];
      f_S    = fsC[ks];
      f_T    = ftC[kzero];
      f_B    = fbC[kb];
      f_NE   = fneC[kzero];
      f_SW   = fswC[ksw];
      f_SE   = fseC[ks];
      f_NW   = fnwC[kw];
      f_TE   = fteC[kzero];
      f_BW   = fbwC[kbw];
      f_BE   = fbeC[kb];
      f_TW   = ftwC[kw];
      f_TN   = ftnC[kzero];
      f_BS   = fbsC[kbs];
      f_BN   = fbnC[kb];
      f_TS   = ftsC[ks];
      f_TNE  = ftneC[kzero];
      f_TSW  = ftswC[ksw];
      f_TSE  = ftseC[ks];
      f_TNW  = ftnwC[kw];
      f_BNE  = fbneC[kb];
      f_BSW  = fbswC[kbsw];
      f_BSE  = fbseC[kbs];
      f_BNW  = fbnwC[kbw];
      ////////////////////////////////////////////////////////////////////////////////
      f27E    =  (D27C.f[dP00])[kzero];//ke
      f27W    =  (D27C.f[dM00])[kw   ];
      f27N    =  (D27C.f[d0P0])[kzero];//kn
      f27S    =  (D27C.f[d0M0])[ks   ];
      f27T    =  (D27C.f[d00P])[kzero];//kt
      f27B    =  (D27C.f[d00M])[kb   ];
      f27NE   =  (D27C.f[dPP0])[kzero];//kne
      f27SW   =  (D27C.f[dMM0])[ksw  ];
      f27SE   =  (D27C.f[dPM0])[ks   ];//kse
      f27NW   =  (D27C.f[dMP0])[kw   ];//knw
      f27TE   =  (D27C.f[dP0P])[kzero];//kte
      f27BW   =  (D27C.f[dM0M])[kbw  ];
      f27BE   =  (D27C.f[dP0M])[kb   ];//kbe
      f27TW   =  (D27C.f[dM0P])[kw   ];//ktw
      f27TN   =  (D27C.f[d0PP])[kzero];//ktn
      f27BS   =  (D27C.f[d0MM])[kbs  ];
      f27BN   =  (D27C.f[d0PM])[kb   ];//kbn
      f27TS   =  (D27C.f[d0MP])[ks   ];//kts
      f27ZERO =  (D27C.f[d000])[kzero];//kzero
      f27TNE   = (D27C.f[dPPP])[kzero];//ktne
      f27TSW   = (D27C.f[dMMP])[ksw  ];//ktsw
      f27TSE   = (D27C.f[dPMP])[ks   ];//ktse
      f27TNW   = (D27C.f[dMPP])[kw   ];//ktnw
      f27BNE   = (D27C.f[dPPM])[kb   ];//kbne
      f27BSW   = (D27C.f[dMMM])[kbsw ];
      f27BSE   = (D27C.f[dPMM])[kbs  ];//kbse
      f27BNW   = (D27C.f[dMPM])[kbw  ];//kbnw

      Conc_C_NEB = f27E + f27W + f27N + f27S + f27T + f27B + f27NE + f27SW + f27SE + f27NW + 
                   f27TE + f27BW + f27BE + f27TW + f27TN + f27BS + f27BN + f27TS + f27ZERO + 
                   f27TNE + f27TSW + f27TSE + f27TNW + f27BNE + f27BSW + f27BSE + f27BNW; 

      vx1  =f_E+f_NE+f_SE+f_TE+f_BE-f_W-f_NW-f_SW-f_TW-f_BW+f_TNE-f_TSW+f_TSE-f_TNW+f_BNE-f_BSW+f_BSE-f_BNW;
      vx2  =f_N+f_NE+f_NW+f_TN+f_BN-f_S-f_SE-f_SW-f_TS-f_BS+f_TNE-f_TSW-f_TSE+f_TNW+f_BNE-f_BSW-f_BSE+f_BNW;
      vx3  =f_T+f_TE+f_TW+f_TN+f_TS-f_B-f_BE-f_BW-f_BN-f_BS+f_TNE+f_TSW+f_TSE+f_TNW-f_BNE-f_BSW-f_BSE-f_BNW;
      Mx   =f27E+f27NE+f27SE+f27TE+f27BE-f27W-f27NW-f27SW-f27TW-f27BW+f27TNE-f27TSW+f27TSE-f27TNW+f27BNE-f27BSW+f27BSE-f27BNW;
      My   =f27N+f27NE+f27NW+f27TN+f27BN-f27S-f27SE-f27SW-f27TS-f27BS+f27TNE-f27TSW-f27TSE+f27TNW+f27BNE-f27BSW-f27BSE+f27BNW;
      Mz   =f27T+f27TE+f27TW+f27TN+f27TS-f27B-f27BE-f27BW-f27BN-f27BS+f27TNE+f27TSW+f27TSE+f27TNW-f27BNE-f27BSW-f27BSE-f27BNW;

      real Diff_Conc_X_CNEB = (Conc_C_NEB * vx1 - Mx) * (c3o1*omegaD_C);
      real Diff_Conc_Y_CNEB = (Conc_C_NEB * vx2 - My) * (c3o1*omegaD_C);
      real Diff_Conc_Z_CNEB = (Conc_C_NEB * vx3 - Mz) * (c3o1*omegaD_C);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //linear
      //real Diff_Conc_XX = zero;
      //real Diff_Conc_YY = zero;
      //real Diff_Conc_ZZ = zero;
      //quadratic
      real Diff_Conc_XX = ((Diff_Conc_X_CNEB + Diff_Conc_X_CSEB + Diff_Conc_X_CNET + Diff_Conc_X_CSET) - (Diff_Conc_X_CNWB + Diff_Conc_X_CSWB + Diff_Conc_X_CNWT + Diff_Conc_X_CSWT)) * c1o4;
      real Diff_Conc_YY = ((Diff_Conc_Y_CNEB + Diff_Conc_Y_CNWB + Diff_Conc_Y_CNET + Diff_Conc_Y_CNWT) - (Diff_Conc_Y_CSEB + Diff_Conc_Y_CSWB + Diff_Conc_Y_CSET + Diff_Conc_Y_CSWT)) * c1o4;
      real Diff_Conc_ZZ = ((Diff_Conc_Z_CSET + Diff_Conc_Z_CSWT + Diff_Conc_Z_CNET + Diff_Conc_Z_CNWT) - (Diff_Conc_Z_CSEB + Diff_Conc_Z_CSWB + Diff_Conc_Z_CNEB + Diff_Conc_Z_CNWB)) * c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

      real dx = c1o4 * (Conc_C_NEB + Conc_C_NET - Conc_C_NWB - Conc_C_NWT + Conc_C_SEB + Conc_C_SET - Conc_C_SWB - Conc_C_SWT);
      real dy = c1o4 * (Conc_C_NEB + Conc_C_NET + Conc_C_NWB + Conc_C_NWT - Conc_C_SEB - Conc_C_SET - Conc_C_SWB - Conc_C_SWT);
      real dz = c1o4 * (-Conc_C_NEB + Conc_C_NET - Conc_C_NWB + Conc_C_NWT - Conc_C_SEB + Conc_C_SET - Conc_C_SWB + Conc_C_SWT);
      real dxx = Diff_Conc_XX * c1o2;
      real dyy = Diff_Conc_YY * c1o2;
      real dzz = Diff_Conc_ZZ * c1o2;
      real dxy = c1o2 * (Conc_C_NEB + Conc_C_NET - Conc_C_NWB - Conc_C_NWT - Conc_C_SEB - Conc_C_SET + Conc_C_SWB + Conc_C_SWT);
      real dyz = c1o2 * (-Conc_C_NEB + Conc_C_NET - Conc_C_NWB + Conc_C_NWT + Conc_C_SEB - Conc_C_SET + Conc_C_SWB - Conc_C_SWT);
      real dxz = c1o2 * (-Conc_C_NEB + Conc_C_NET + Conc_C_NWB - Conc_C_NWT - Conc_C_SEB + Conc_C_SET + Conc_C_SWB - Conc_C_SWT);
      real dxyz = -Conc_C_NEB + Conc_C_NET + Conc_C_NWB - Conc_C_NWT + Conc_C_SEB - Conc_C_SET - Conc_C_SWB + Conc_C_SWT;
      real d0 = c1o8 * (-c2o1 * dxx - c2o1 * dyy - c2o1 * dzz + Conc_C_NEB + Conc_C_NET + Conc_C_NWB + Conc_C_NWT + Conc_C_SEB + Conc_C_SET + Conc_C_SWB + Conc_C_SWT);

     ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //
      // offset for refinement into the wall
      //
      // X------X
      // |      | x---x    
      // |   ---+-+-> |    ----> off-vector
      // |      | x---x 
      // X------X   
      //            
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      d0 = d0 + xoff * dx + yoff * dy + zoff * dz + xoff_sq * dxx + yoff_sq * dyy + zoff_sq * dzz + xoff*yoff*dxy + xoff*zoff*dxz + yoff*zoff*dyz;
      dx = dx + c2o1 * xoff * dxx + yoff * dxy + zoff * dxz;
      dy = dy + c2o1 * yoff * dyy + xoff * dxy + zoff * dyz;
      dz = dz + c2o1 * zoff * dzz + xoff * dxz + yoff * dyz;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position SWB -0.25, -0.25, -0.25
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      x = -c1o4;
      y = -c1o4;
      z = -c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 0
      k0zero= posFSWB[k];
      k0w   = neighborFX[k0zero];
      k0s   = neighborFY[k0zero];
      k0b   = neighborFZ[k0zero];
      k0sw  = neighborFY[k0w];
      k0bw  = neighborFZ[k0w];
      k0bs  = neighborFZ[k0s];
      k0bsw = neighborFZ[k0sw];
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 
      kzero= k0zero;
      kw   = k0w;   
      ks   = k0s;   
      kb   = k0b;   
      ksw  = k0sw;  
      kbw  = k0bw;  
      kbs  = k0bs;  
      kbsw = k0bsw; 
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      vx1=feF[kzero]+fneF[kzero]+fseF[ks]+fteF[kzero]+fbeF[kb]-fwF[kw]-fnwF[kw]-fswF[ksw]-ftwF[kw]-fbwF[kbw]+ftneF[kzero]-ftswF[ksw]+ftseF[ks]-ftnwF[kw]+fbneF[kb]-fbswF[kbsw]+fbseF[kbs]-fbnwF[kbw];
      vx2=fnF[kzero]+fneF[kzero]+fnwF[kw]+ftnF[kzero]+fbnF[kb]-fsF[ks]-fseF[ks]-fswF[ksw]-ftsF[ks]-fbsF[kbs]+ftneF[kzero]-ftswF[ksw]-ftseF[ks]+ftnwF[kw]+fbneF[kb]-fbswF[kbsw]-fbseF[kbs]+fbnwF[kbw];
      vx3=ftF[kzero]+fteF[kzero]+ftwF[kw]+ftnF[kzero]+ftsF[ks]-fbF[kb]-fbeF[kb]-fbwF[kbw]-fbnF[kb]-fbsF[kbs]+ftneF[kzero]+ftswF[ksw]+ftseF[ks]+ftnwF[kw]-fbneF[kb]-fbswF[kbsw]-fbseF[kbs]-fbnwF[kbw];

      real Conc_F = d0 + dx*x + dy*y + dz*z + dxx*x*x + dyy*y*y + dzz*z*z + dxy*x*y +  dxz*x*z + dyz*y*z + dxyz*x*y*z;

      real Diff_Conc_X = dx + x * dxx + y * dxy + z * dxz + y * z * dxyz;
      real Diff_Conc_Y = dy + y * dyy + x * dxy + z * dyz + x * z * dxyz;
      real Diff_Conc_Z = dz + z * dzz + x * dxz + y * dyz + x * y * dxyz;

      Mx = Conc_F*vx1-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_X;
      My = Conc_F*vx2-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Y;
      Mz = Conc_F*vx3-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Z;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27F.f[d000])[kzero] =   c8o27* Conc_F*(c1o1-cu_sq);
      (D27F.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_F*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27F.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_F*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27F.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_F*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27F.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_F*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27F.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_F*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27F.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_F*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27F.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_F*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27F.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_F*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27F.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_F*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27F.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_F*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27F.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_F*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27F.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_F*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27F.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_F*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27F.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_F*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27F.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_F*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27F.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_F*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27F.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_F*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27F.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_F*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27F.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27F.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27F.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27F.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27F.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27F.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27F.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27F.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));








      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position SWT -0.25, -0.25, 0.25
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      x = -c1o4;
      y = -c1o4;
      z =  c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kb;
      kw   = kbw;   
      ks   = kbs;   
      kb   = neighborFZ[kb];   
      ksw  = kbsw;  
      kbw  = neighborFZ[kbw];  
      kbs  = neighborFZ[kbs];  
      kbsw = neighborFZ[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      vx1=feF[kzero]+fneF[kzero]+fseF[ks]+fteF[kzero]+fbeF[kb]-fwF[kw]-fnwF[kw]-fswF[ksw]-ftwF[kw]-fbwF[kbw]+ftneF[kzero]-ftswF[ksw]+ftseF[ks]-ftnwF[kw]+fbneF[kb]-fbswF[kbsw]+fbseF[kbs]-fbnwF[kbw];
      vx2=fnF[kzero]+fneF[kzero]+fnwF[kw]+ftnF[kzero]+fbnF[kb]-fsF[ks]-fseF[ks]-fswF[ksw]-ftsF[ks]-fbsF[kbs]+ftneF[kzero]-ftswF[ksw]-ftseF[ks]+ftnwF[kw]+fbneF[kb]-fbswF[kbsw]-fbseF[kbs]+fbnwF[kbw];
      vx3=ftF[kzero]+fteF[kzero]+ftwF[kw]+ftnF[kzero]+ftsF[ks]-fbF[kb]-fbeF[kb]-fbwF[kbw]-fbnF[kb]-fbsF[kbs]+ftneF[kzero]+ftswF[ksw]+ftseF[ks]+ftnwF[kw]-fbneF[kb]-fbswF[kbsw]-fbseF[kbs]-fbnwF[kbw];

      Conc_F = d0 + dx*x + dy*y + dz*z + dxx*x*x + dyy*y*y + dzz*z*z + dxy*x*y +  dxz*x*z + dyz*y*z + dxyz*x*y*z;

      Diff_Conc_X = dx + x * dxx + y * dxy + z * dxz + y * z * dxyz;
      Diff_Conc_Y = dy + y * dyy + x * dxy + z * dyz + x * z * dxyz;
      Diff_Conc_Z = dz + z * dzz + x * dxz + y * dyz + x * y * dxyz;

      Mx = Conc_F*vx1-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_X;
      My = Conc_F*vx2-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Y;
      Mz = Conc_F*vx3-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Z;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27F.f[d000])[kzero] =   c8o27* Conc_F*(c1o1-cu_sq);
      (D27F.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_F*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27F.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_F*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27F.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_F*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27F.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_F*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27F.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_F*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27F.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_F*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27F.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_F*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27F.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_F*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27F.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_F*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27F.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_F*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27F.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_F*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27F.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_F*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27F.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_F*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27F.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_F*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27F.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_F*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27F.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_F*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27F.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_F*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27F.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_F*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27F.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27F.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27F.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27F.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27F.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27F.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27F.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27F.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));








      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position SET 0.25, -0.25, 0.25
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      x =  c1o4;
      y = -c1o4;
      z =  c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kw;
      kw   = neighborFX[kw];   
      ks   = ksw;   
      kb   = kbw;   
      ksw  = neighborFX[ksw];  
      kbw  = neighborFX[kbw];  
      kbs  = kbsw;  
      kbsw = neighborFX[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      vx1=feF[kzero]+fneF[kzero]+fseF[ks]+fteF[kzero]+fbeF[kb]-fwF[kw]-fnwF[kw]-fswF[ksw]-ftwF[kw]-fbwF[kbw]+ftneF[kzero]-ftswF[ksw]+ftseF[ks]-ftnwF[kw]+fbneF[kb]-fbswF[kbsw]+fbseF[kbs]-fbnwF[kbw];
      vx2=fnF[kzero]+fneF[kzero]+fnwF[kw]+ftnF[kzero]+fbnF[kb]-fsF[ks]-fseF[ks]-fswF[ksw]-ftsF[ks]-fbsF[kbs]+ftneF[kzero]-ftswF[ksw]-ftseF[ks]+ftnwF[kw]+fbneF[kb]-fbswF[kbsw]-fbseF[kbs]+fbnwF[kbw];
      vx3=ftF[kzero]+fteF[kzero]+ftwF[kw]+ftnF[kzero]+ftsF[ks]-fbF[kb]-fbeF[kb]-fbwF[kbw]-fbnF[kb]-fbsF[kbs]+ftneF[kzero]+ftswF[ksw]+ftseF[ks]+ftnwF[kw]-fbneF[kb]-fbswF[kbsw]-fbseF[kbs]-fbnwF[kbw];

      Conc_F = d0 + dx*x + dy*y + dz*z + dxx*x*x + dyy*y*y + dzz*z*z + dxy*x*y +  dxz*x*z + dyz*y*z + dxyz*x*y*z;

      Diff_Conc_X = dx + x * dxx + y * dxy + z * dxz + y * z * dxyz;
      Diff_Conc_Y = dy + y * dyy + x * dxy + z * dyz + x * z * dxyz;
      Diff_Conc_Z = dz + z * dzz + x * dxz + y * dyz + x * y * dxyz;

      Mx = Conc_F*vx1-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_X;
      My = Conc_F*vx2-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Y;
      Mz = Conc_F*vx3-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Z;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27F.f[d000])[kzero] =   c8o27* Conc_F*(c1o1-cu_sq);
      (D27F.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_F*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27F.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_F*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27F.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_F*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27F.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_F*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27F.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_F*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27F.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_F*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27F.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_F*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27F.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_F*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27F.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_F*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27F.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_F*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27F.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_F*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27F.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_F*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27F.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_F*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27F.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_F*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27F.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_F*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27F.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_F*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27F.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_F*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27F.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_F*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27F.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27F.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27F.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27F.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27F.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27F.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27F.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27F.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));








      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position SEB 0.25, -0.25, -0.25
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      x =  c1o4;
      y = -c1o4;
      z = -c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 
      kb   = kzero;   
      kbw  = kw;  
      kbs  = ks;  
      kbsw = ksw; 
      kzero= k0w;
      kw   = neighborFX[k0w];   
      ks   = k0sw;   
      ksw  = neighborFX[k0sw];  
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      vx1=feF[kzero]+fneF[kzero]+fseF[ks]+fteF[kzero]+fbeF[kb]-fwF[kw]-fnwF[kw]-fswF[ksw]-ftwF[kw]-fbwF[kbw]+ftneF[kzero]-ftswF[ksw]+ftseF[ks]-ftnwF[kw]+fbneF[kb]-fbswF[kbsw]+fbseF[kbs]-fbnwF[kbw];
      vx2=fnF[kzero]+fneF[kzero]+fnwF[kw]+ftnF[kzero]+fbnF[kb]-fsF[ks]-fseF[ks]-fswF[ksw]-ftsF[ks]-fbsF[kbs]+ftneF[kzero]-ftswF[ksw]-ftseF[ks]+ftnwF[kw]+fbneF[kb]-fbswF[kbsw]-fbseF[kbs]+fbnwF[kbw];
      vx3=ftF[kzero]+fteF[kzero]+ftwF[kw]+ftnF[kzero]+ftsF[ks]-fbF[kb]-fbeF[kb]-fbwF[kbw]-fbnF[kb]-fbsF[kbs]+ftneF[kzero]+ftswF[ksw]+ftseF[ks]+ftnwF[kw]-fbneF[kb]-fbswF[kbsw]-fbseF[kbs]-fbnwF[kbw];

      Conc_F = d0 + dx*x + dy*y + dz*z + dxx*x*x + dyy*y*y + dzz*z*z + dxy*x*y +  dxz*x*z + dyz*y*z + dxyz*x*y*z;

      Diff_Conc_X = dx + x * dxx + y * dxy + z * dxz + y * z * dxyz;
      Diff_Conc_Y = dy + y * dyy + x * dxy + z * dyz + x * z * dxyz;
      Diff_Conc_Z = dz + z * dzz + x * dxz + y * dyz + x * y * dxyz;

      Mx = Conc_F*vx1-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_X;
      My = Conc_F*vx2-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Y;
      Mz = Conc_F*vx3-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Z;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27F.f[d000])[kzero] =   c8o27* Conc_F*(c1o1-cu_sq);
      (D27F.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_F*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27F.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_F*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27F.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_F*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27F.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_F*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27F.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_F*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27F.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_F*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27F.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_F*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27F.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_F*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27F.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_F*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27F.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_F*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27F.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_F*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27F.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_F*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27F.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_F*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27F.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_F*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27F.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_F*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27F.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_F*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27F.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_F*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27F.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_F*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27F.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27F.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27F.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27F.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27F.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27F.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27F.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27F.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));








      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position NWB -0.25, 0.25, -0.25
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      x = -c1o4;
      y =  c1o4;
      z = -c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 0
      k0zero= k0s;
      k0w   = k0sw;
      k0s   = neighborFY[k0s];
      k0b   = k0bs;
      k0sw  = neighborFY[k0sw];
      k0bw  = k0bsw;
      k0bs  = neighborFY[k0bs];
      k0bsw = neighborFY[k0bsw];
      //////////////////////////////////////////////////////////////////////////
      //index 
      kzero= k0zero;
      kw   = k0w;   
      ks   = k0s;   
      kb   = k0b;   
      ksw  = k0sw;  
      kbw  = k0bw;  
      kbs  = k0bs;  
      kbsw = k0bsw; 
      ////////////////////////////////////////////////////////////////////////////////
      vx1=feF[kzero]+fneF[kzero]+fseF[ks]+fteF[kzero]+fbeF[kb]-fwF[kw]-fnwF[kw]-fswF[ksw]-ftwF[kw]-fbwF[kbw]+ftneF[kzero]-ftswF[ksw]+ftseF[ks]-ftnwF[kw]+fbneF[kb]-fbswF[kbsw]+fbseF[kbs]-fbnwF[kbw];
      vx2=fnF[kzero]+fneF[kzero]+fnwF[kw]+ftnF[kzero]+fbnF[kb]-fsF[ks]-fseF[ks]-fswF[ksw]-ftsF[ks]-fbsF[kbs]+ftneF[kzero]-ftswF[ksw]-ftseF[ks]+ftnwF[kw]+fbneF[kb]-fbswF[kbsw]-fbseF[kbs]+fbnwF[kbw];
      vx3=ftF[kzero]+fteF[kzero]+ftwF[kw]+ftnF[kzero]+ftsF[ks]-fbF[kb]-fbeF[kb]-fbwF[kbw]-fbnF[kb]-fbsF[kbs]+ftneF[kzero]+ftswF[ksw]+ftseF[ks]+ftnwF[kw]-fbneF[kb]-fbswF[kbsw]-fbseF[kbs]-fbnwF[kbw];

      Conc_F = d0 + dx*x + dy*y + dz*z + dxx*x*x + dyy*y*y + dzz*z*z + dxy*x*y +  dxz*x*z + dyz*y*z + dxyz*x*y*z;

      Diff_Conc_X = dx + x * dxx + y * dxy + z * dxz + y * z * dxyz;
      Diff_Conc_Y = dy + y * dyy + x * dxy + z * dyz + x * z * dxyz;
      Diff_Conc_Z = dz + z * dzz + x * dxz + y * dyz + x * y * dxyz;

      Mx = Conc_F*vx1-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_X;
      My = Conc_F*vx2-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Y;
      Mz = Conc_F*vx3-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Z;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27F.f[d000])[kzero] =   c8o27* Conc_F*(c1o1-cu_sq);
      (D27F.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_F*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27F.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_F*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27F.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_F*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27F.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_F*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27F.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_F*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27F.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_F*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27F.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_F*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27F.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_F*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27F.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_F*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27F.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_F*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27F.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_F*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27F.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_F*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27F.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_F*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27F.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_F*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27F.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_F*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27F.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_F*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27F.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_F*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27F.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_F*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27F.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27F.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27F.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27F.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27F.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27F.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27F.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27F.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));








      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position NWT -0.25, 0.25, 0.25
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      x = -c1o4;
      y =  c1o4;
      z =  c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kb;
      kw   = kbw;   
      ks   = kbs;   
      kb   = neighborFZ[kb];   
      ksw  = kbsw;  
      kbw  = neighborFZ[kbw];  
      kbs  = neighborFZ[kbs];  
      kbsw = neighborFZ[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      vx1=feF[kzero]+fneF[kzero]+fseF[ks]+fteF[kzero]+fbeF[kb]-fwF[kw]-fnwF[kw]-fswF[ksw]-ftwF[kw]-fbwF[kbw]+ftneF[kzero]-ftswF[ksw]+ftseF[ks]-ftnwF[kw]+fbneF[kb]-fbswF[kbsw]+fbseF[kbs]-fbnwF[kbw];
      vx2=fnF[kzero]+fneF[kzero]+fnwF[kw]+ftnF[kzero]+fbnF[kb]-fsF[ks]-fseF[ks]-fswF[ksw]-ftsF[ks]-fbsF[kbs]+ftneF[kzero]-ftswF[ksw]-ftseF[ks]+ftnwF[kw]+fbneF[kb]-fbswF[kbsw]-fbseF[kbs]+fbnwF[kbw];
      vx3=ftF[kzero]+fteF[kzero]+ftwF[kw]+ftnF[kzero]+ftsF[ks]-fbF[kb]-fbeF[kb]-fbwF[kbw]-fbnF[kb]-fbsF[kbs]+ftneF[kzero]+ftswF[ksw]+ftseF[ks]+ftnwF[kw]-fbneF[kb]-fbswF[kbsw]-fbseF[kbs]-fbnwF[kbw];

      Conc_F = d0 + dx*x + dy*y + dz*z + dxx*x*x + dyy*y*y + dzz*z*z + dxy*x*y +  dxz*x*z + dyz*y*z + dxyz*x*y*z;

      Diff_Conc_X = dx + x * dxx + y * dxy + z * dxz + y * z * dxyz;
      Diff_Conc_Y = dy + y * dyy + x * dxy + z * dyz + x * z * dxyz;
      Diff_Conc_Z = dz + z * dzz + x * dxz + y * dyz + x * y * dxyz;

      Mx = Conc_F*vx1-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_X;
      My = Conc_F*vx2-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Y;
      Mz = Conc_F*vx3-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Z;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27F.f[d000])[kzero] =   c8o27* Conc_F*(c1o1-cu_sq);
      (D27F.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_F*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27F.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_F*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27F.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_F*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27F.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_F*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27F.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_F*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27F.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_F*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27F.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_F*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27F.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_F*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27F.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_F*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27F.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_F*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27F.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_F*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27F.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_F*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27F.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_F*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27F.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_F*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27F.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_F*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27F.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_F*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27F.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_F*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27F.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_F*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27F.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27F.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27F.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27F.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27F.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27F.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27F.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27F.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));








      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position NET 0.25, 0.25, 0.25
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      x = c1o4;
      y = c1o4;
      z = c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 
      kzero= kw;
      kw   = neighborFX[kw];   
      ks   = ksw;   
      kb   = kbw;   
      ksw  = neighborFX[ksw];  
      kbw  = neighborFX[kbw];  
      kbs  = kbsw;  
      kbsw = neighborFX[kbsw]; 
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      vx1=feF[kzero]+fneF[kzero]+fseF[ks]+fteF[kzero]+fbeF[kb]-fwF[kw]-fnwF[kw]-fswF[ksw]-ftwF[kw]-fbwF[kbw]+ftneF[kzero]-ftswF[ksw]+ftseF[ks]-ftnwF[kw]+fbneF[kb]-fbswF[kbsw]+fbseF[kbs]-fbnwF[kbw];
      vx2=fnF[kzero]+fneF[kzero]+fnwF[kw]+ftnF[kzero]+fbnF[kb]-fsF[ks]-fseF[ks]-fswF[ksw]-ftsF[ks]-fbsF[kbs]+ftneF[kzero]-ftswF[ksw]-ftseF[ks]+ftnwF[kw]+fbneF[kb]-fbswF[kbsw]-fbseF[kbs]+fbnwF[kbw];
      vx3=ftF[kzero]+fteF[kzero]+ftwF[kw]+ftnF[kzero]+ftsF[ks]-fbF[kb]-fbeF[kb]-fbwF[kbw]-fbnF[kb]-fbsF[kbs]+ftneF[kzero]+ftswF[ksw]+ftseF[ks]+ftnwF[kw]-fbneF[kb]-fbswF[kbsw]-fbseF[kbs]-fbnwF[kbw];

      Conc_F = d0 + dx*x + dy*y + dz*z + dxx*x*x + dyy*y*y + dzz*z*z + dxy*x*y +  dxz*x*z + dyz*y*z + dxyz*x*y*z;

      Diff_Conc_X = dx + x * dxx + y * dxy + z * dxz + y * z * dxyz;
      Diff_Conc_Y = dy + y * dyy + x * dxy + z * dyz + x * z * dxyz;
      Diff_Conc_Z = dz + z * dzz + x * dxz + y * dyz + x * y * dxyz;

      Mx = Conc_F*vx1-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_X;
      My = Conc_F*vx2-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Y;
      Mz = Conc_F*vx3-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Z;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27F.f[d000])[kzero] =   c8o27* Conc_F*(c1o1-cu_sq);
      (D27F.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_F*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27F.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_F*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27F.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_F*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27F.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_F*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27F.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_F*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27F.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_F*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27F.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_F*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27F.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_F*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27F.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_F*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27F.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_F*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27F.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_F*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27F.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_F*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27F.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_F*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27F.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_F*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27F.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_F*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27F.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_F*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27F.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_F*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27F.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_F*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27F.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27F.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27F.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27F.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27F.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27F.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27F.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27F.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));








      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Position NEB 0.25, 0.25, -0.25
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      x =  c1o4;
      y =  c1o4;
      z = -c1o4;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //index 
      kb   = kzero;   
      kbw  = kw;  
      kbs  = ks;  
      kbsw = ksw; 
      kzero= k0w;
      kw   = neighborFX[k0w];   
      ks   = k0sw;   
      ksw  = neighborFX[k0sw];  
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      vx1=feF[kzero]+fneF[kzero]+fseF[ks]+fteF[kzero]+fbeF[kb]-fwF[kw]-fnwF[kw]-fswF[ksw]-ftwF[kw]-fbwF[kbw]+ftneF[kzero]-ftswF[ksw]+ftseF[ks]-ftnwF[kw]+fbneF[kb]-fbswF[kbsw]+fbseF[kbs]-fbnwF[kbw];
      vx2=fnF[kzero]+fneF[kzero]+fnwF[kw]+ftnF[kzero]+fbnF[kb]-fsF[ks]-fseF[ks]-fswF[ksw]-ftsF[ks]-fbsF[kbs]+ftneF[kzero]-ftswF[ksw]-ftseF[ks]+ftnwF[kw]+fbneF[kb]-fbswF[kbsw]-fbseF[kbs]+fbnwF[kbw];
      vx3=ftF[kzero]+fteF[kzero]+ftwF[kw]+ftnF[kzero]+ftsF[ks]-fbF[kb]-fbeF[kb]-fbwF[kbw]-fbnF[kb]-fbsF[kbs]+ftneF[kzero]+ftswF[ksw]+ftseF[ks]+ftnwF[kw]-fbneF[kb]-fbswF[kbsw]-fbseF[kbs]-fbnwF[kbw];

      Conc_F = d0 + dx*x + dy*y + dz*z + dxx*x*x + dyy*y*y + dzz*z*z + dxy*x*y +  dxz*x*z + dyz*y*z + dxyz*x*y*z;

      Diff_Conc_X = dx + x * dxx + y * dxy + z * dxz + y * z * dxyz;
      Diff_Conc_Y = dy + y * dyy + x * dxy + z * dyz + x * z * dxyz;
      Diff_Conc_Z = dz + z * dzz + x * dxz + y * dyz + x * y * dxyz;

      Mx = Conc_F*vx1-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_X;
      My = Conc_F*vx2-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Y;
      Mz = Conc_F*vx3-(c1o1)/(c3o1*omegaD_F)*c1o2*Diff_Conc_Z;

      cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      (D27F.f[d000])[kzero] =   c8o27* Conc_F*(c1o1-cu_sq);
      (D27F.f[dP00])[kzero] =   c2o27* (c3o1*( Mx        )+Conc_F*(c1o1+c9o2*( vx1        )*( vx1        )-cu_sq));
      (D27F.f[dM00])[kw   ] =   c2o27* (c3o1*(-Mx        )+Conc_F*(c1o1+c9o2*(-vx1        )*(-vx1        )-cu_sq));
      (D27F.f[d0P0])[kzero] =   c2o27* (c3o1*(     My    )+Conc_F*(c1o1+c9o2*(     vx2    )*(     vx2    )-cu_sq));
      (D27F.f[d0M0])[ks   ] =   c2o27* (c3o1*(    -My    )+Conc_F*(c1o1+c9o2*(    -vx2    )*(    -vx2    )-cu_sq));
      (D27F.f[d00P])[kzero] =   c2o27* (c3o1*(         Mz)+Conc_F*(c1o1+c9o2*(         vx3)*(         vx3)-cu_sq));
      (D27F.f[d00M])[kb   ] =   c2o27* (c3o1*(        -Mz)+Conc_F*(c1o1+c9o2*(        -vx3)*(        -vx3)-cu_sq));
      (D27F.f[dPP0])[kzero] =   c1o54* (c3o1*( Mx +My    )+Conc_F*(c1o1+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq));
      (D27F.f[dMM0])[ksw  ] =   c1o54* (c3o1*(-Mx -My    )+Conc_F*(c1o1+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq));
      (D27F.f[dPM0])[ks   ] =   c1o54* (c3o1*( Mx -My    )+Conc_F*(c1o1+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq));
      (D27F.f[dMP0])[kw   ] =   c1o54* (c3o1*(-Mx +My    )+Conc_F*(c1o1+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq));
      (D27F.f[dP0P])[kzero] =   c1o54* (c3o1*( Mx     +Mz)+Conc_F*(c1o1+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq));
      (D27F.f[dM0M])[kbw  ] =   c1o54* (c3o1*(-Mx     -Mz)+Conc_F*(c1o1+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq));
      (D27F.f[dP0M])[kb   ] =   c1o54* (c3o1*( Mx     -Mz)+Conc_F*(c1o1+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq));
      (D27F.f[dM0P])[kw   ] =   c1o54* (c3o1*(-Mx     +Mz)+Conc_F*(c1o1+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq));
      (D27F.f[d0PP])[kzero] =   c1o54* (c3o1*(     My +Mz)+Conc_F*(c1o1+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq));
      (D27F.f[d0MM])[kbs  ] =   c1o54* (c3o1*(    -My -Mz)+Conc_F*(c1o1+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq));
      (D27F.f[d0PM])[kb   ] =   c1o54* (c3o1*(     My -Mz)+Conc_F*(c1o1+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq));
      (D27F.f[d0MP])[ks   ] =   c1o54* (c3o1*(    -My +Mz)+Conc_F*(c1o1+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq));
      (D27F.f[dPPP])[kzero] =   c1o216*(c3o1*( Mx +My +Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq));
      (D27F.f[dMMM])[kbsw ] =   c1o216*(c3o1*(-Mx -My -Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq));
      (D27F.f[dPPM])[kb   ] =   c1o216*(c3o1*( Mx +My -Mz)+Conc_F*(c1o1+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq));
      (D27F.f[dMMP])[ksw  ] =   c1o216*(c3o1*(-Mx -My +Mz)+Conc_F*(c1o1+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq));
      (D27F.f[dPMP])[ks   ] =   c1o216*(c3o1*( Mx -My +Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq));
      (D27F.f[dMPM])[kbw  ] =   c1o216*(c3o1*(-Mx +My -Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq));
      (D27F.f[dPMM])[kbs  ] =   c1o216*(c3o1*( Mx -My -Mz)+Conc_F*(c1o1+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq));
      (D27F.f[dMPP])[kw   ] =   c1o216*(c3o1*(-Mx +My +Mz)+Conc_F*(c1o1+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq));
   }
}
