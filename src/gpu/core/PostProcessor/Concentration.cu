#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  \
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \author Martin Schoenherr
//=======================================================================================
#include "Concentration.cuh"

#include <hip/hip_runtime_api.h>

#include <cuda_helper/CudaGrid.h>

#include <lbm/constants/D3Q27.h>

#include <basics/constants/NumericConstants.h>

#include "LBM/LB.h"

using namespace vf::basics::constant;
using namespace vf::lbm::dir;

////////////////////////////////////////////////////////////////////////////////
__global__ void CalcConc27(
    real* concentration,
    uint* typeOfGridNode,
    uint* neighborX,
    uint* neighborY,
    uint* neighborZ,
    unsigned long long numberOfLBnodes,
    real* distributionsAD,
    bool isEvenTimestep)
{
   //////////////////////////////////////////////////////////////////////////
   //! The velocity boundary condition is executed in the following steps
   //!
   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned  x = threadIdx.x;  // global x-index
   const unsigned  y = blockIdx.x;   // global y-index
   const unsigned  z = blockIdx.y;   // global z-index

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   //////////////////////////////////////////////////////////////////////////
   // run for all indices in size_Mat and fluid nodes
   if ((k < numberOfLBnodes) && (typeOfGridNode[k] == GEO_FLUID))
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 distAD;
      if (isEvenTimestep)
      {
         distAD.f[dP00] = &distributionsAD[dP00 * numberOfLBnodes];
         distAD.f[dM00] = &distributionsAD[dM00 * numberOfLBnodes];
         distAD.f[d0P0] = &distributionsAD[d0P0 * numberOfLBnodes];
         distAD.f[d0M0] = &distributionsAD[d0M0 * numberOfLBnodes];
         distAD.f[d00P] = &distributionsAD[d00P * numberOfLBnodes];
         distAD.f[d00M] = &distributionsAD[d00M * numberOfLBnodes];
         distAD.f[dPP0] = &distributionsAD[dPP0 * numberOfLBnodes];
         distAD.f[dMM0] = &distributionsAD[dMM0 * numberOfLBnodes];
         distAD.f[dPM0] = &distributionsAD[dPM0 * numberOfLBnodes];
         distAD.f[dMP0] = &distributionsAD[dMP0 * numberOfLBnodes];
         distAD.f[dP0P] = &distributionsAD[dP0P * numberOfLBnodes];
         distAD.f[dM0M] = &distributionsAD[dM0M * numberOfLBnodes];
         distAD.f[dP0M] = &distributionsAD[dP0M * numberOfLBnodes];
         distAD.f[dM0P] = &distributionsAD[dM0P * numberOfLBnodes];
         distAD.f[d0PP] = &distributionsAD[d0PP * numberOfLBnodes];
         distAD.f[d0MM] = &distributionsAD[d0MM * numberOfLBnodes];
         distAD.f[d0PM] = &distributionsAD[d0PM * numberOfLBnodes];
         distAD.f[d0MP] = &distributionsAD[d0MP * numberOfLBnodes];
         distAD.f[d000] = &distributionsAD[d000 * numberOfLBnodes];
         distAD.f[dPPP] = &distributionsAD[dPPP * numberOfLBnodes];
         distAD.f[dMMP] = &distributionsAD[dMMP * numberOfLBnodes];
         distAD.f[dPMP] = &distributionsAD[dPMP * numberOfLBnodes];
         distAD.f[dMPP] = &distributionsAD[dMPP * numberOfLBnodes];
         distAD.f[dPPM] = &distributionsAD[dPPM * numberOfLBnodes];
         distAD.f[dMMM] = &distributionsAD[dMMM * numberOfLBnodes];
         distAD.f[dPMM] = &distributionsAD[dPMM * numberOfLBnodes];
         distAD.f[dMPM] = &distributionsAD[dMPM * numberOfLBnodes];
      }
      else
      {
         distAD.f[dM00] = &distributionsAD[dP00 * numberOfLBnodes];
         distAD.f[dP00] = &distributionsAD[dM00 * numberOfLBnodes];
         distAD.f[d0M0] = &distributionsAD[d0P0 * numberOfLBnodes];
         distAD.f[d0P0] = &distributionsAD[d0M0 * numberOfLBnodes];
         distAD.f[d00M] = &distributionsAD[d00P * numberOfLBnodes];
         distAD.f[d00P] = &distributionsAD[d00M * numberOfLBnodes];
         distAD.f[dMM0] = &distributionsAD[dPP0 * numberOfLBnodes];
         distAD.f[dPP0] = &distributionsAD[dMM0 * numberOfLBnodes];
         distAD.f[dMP0] = &distributionsAD[dPM0 * numberOfLBnodes];
         distAD.f[dPM0] = &distributionsAD[dMP0 * numberOfLBnodes];
         distAD.f[dM0M] = &distributionsAD[dP0P * numberOfLBnodes];
         distAD.f[dP0P] = &distributionsAD[dM0M * numberOfLBnodes];
         distAD.f[dM0P] = &distributionsAD[dP0M * numberOfLBnodes];
         distAD.f[dP0M] = &distributionsAD[dM0P * numberOfLBnodes];
         distAD.f[d0MM] = &distributionsAD[d0PP * numberOfLBnodes];
         distAD.f[d0PP] = &distributionsAD[d0MM * numberOfLBnodes];
         distAD.f[d0MP] = &distributionsAD[d0PM * numberOfLBnodes];
         distAD.f[d0PM] = &distributionsAD[d0MP * numberOfLBnodes];
         distAD.f[d000] = &distributionsAD[d000 * numberOfLBnodes];
         distAD.f[dPPP] = &distributionsAD[dMMM * numberOfLBnodes];
         distAD.f[dMMP] = &distributionsAD[dPPM * numberOfLBnodes];
         distAD.f[dPMP] = &distributionsAD[dMPM * numberOfLBnodes];
         distAD.f[dMPP] = &distributionsAD[dPMM * numberOfLBnodes];
         distAD.f[dPPM] = &distributionsAD[dMMP * numberOfLBnodes];
         distAD.f[dMMM] = &distributionsAD[dPPP * numberOfLBnodes];
         distAD.f[dPMM] = &distributionsAD[dMPP * numberOfLBnodes];
         distAD.f[dMPM] = &distributionsAD[dPMP * numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      uint ke   = k;
      uint kw   = neighborX[k];
      uint kn   = k;
      uint ks   = neighborY[k];
      uint kt   = k;
      uint kb   = neighborZ[k];
      uint ksw  = neighborY[kw];
      uint kne  = k;
      uint kse  = ks;
      uint knw  = kw;
      uint kbw  = neighborZ[kw];
      uint kte  = k;
      uint kbe  = kb;
      uint ktw  = kw;
      uint kbs  = neighborZ[ks];
      uint ktn  = k;
      uint kbn  = kb;
      uint kts  = ks;
      uint ktse = ks;
      uint kbnw = kbw;
      uint ktnw = kw;
      uint kbse = kbs;
      uint ktsw = ksw;
      uint kbne = kb;
      uint ktne = k;
      uint kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real mfcbb = (distAD.f[dP00])[ke  ];
      real mfabb = (distAD.f[dM00])[kw  ];
      real mfbcb = (distAD.f[d0P0])[kn  ];
      real mfbab = (distAD.f[d0M0])[ks  ];
      real mfbbc = (distAD.f[d00P])[kt  ];
      real mfbba = (distAD.f[d00M])[kb  ];
      real mfccb = (distAD.f[dPP0])[kne ];
      real mfaab = (distAD.f[dMM0])[ksw ];
      real mfcab = (distAD.f[dPM0])[kse ];
      real mfacb = (distAD.f[dMP0])[knw ];
      real mfcbc = (distAD.f[dP0P])[kte ];
      real mfaba = (distAD.f[dM0M])[kbw ];
      real mfcba = (distAD.f[dP0M])[kbe ];
      real mfabc = (distAD.f[dM0P])[ktw ];
      real mfbcc = (distAD.f[d0PP])[ktn ];
      real mfbaa = (distAD.f[d0MM])[kbs ];
      real mfbca = (distAD.f[d0PM])[kbn ];
      real mfbac = (distAD.f[d0MP])[kts ];
      real mfbbb = (distAD.f[d000])[k   ];
      real mfccc = (distAD.f[dPPP])[ktne];
      real mfaac = (distAD.f[dMMP])[ktsw];
      real mfcac = (distAD.f[dPMP])[ktse];
      real mfacc = (distAD.f[dMPP])[ktnw];
      real mfcca = (distAD.f[dPPM])[kbne];
      real mfaaa = (distAD.f[dMMM])[kbsw];
      real mfcaa = (distAD.f[dPMM])[kbse];
      real mfaca = (distAD.f[dMPM])[kbnw];
      //////////////////////////////////////////////////////////////////////////
      //! - Calculate concentration using pyramid summation for low round-off errors as in Eq. (J1)-(J3) \ref
      //! <a href="https://doi.org/10.1016/j.camwa.2015.05.001"><b>[ M. Geier et al. (2015), DOI:10.1016/j.camwa.2015.05.001 ]</b></a>
      //!
      concentration[k] =
       ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa)   + (mfaac + mfcca))) +
          (((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba)   + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
           ((mfabb + mfcbb) + (mfbab + mfbcb)  +  (mfbba + mfbbc))) +  mfbbb;

   }
}



















////////////////////////////////////////////////////////////////////////////////
__global__ void CalcConc7( real* Conc,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned long long numberOfLBnodes,
                                          real* DD7,
                                          bool isEvenTimestep)
{
   Distributions7 D7;
   if (isEvenTimestep==true)
   {
      D7.f[0] = &DD7[0*numberOfLBnodes];
      D7.f[1] = &DD7[1*numberOfLBnodes];
      D7.f[2] = &DD7[2*numberOfLBnodes];
      D7.f[3] = &DD7[3*numberOfLBnodes];
      D7.f[4] = &DD7[4*numberOfLBnodes];
      D7.f[5] = &DD7[5*numberOfLBnodes];
      D7.f[6] = &DD7[6*numberOfLBnodes];
   } 
   else
   {
      D7.f[0] = &DD7[0*numberOfLBnodes];
      D7.f[2] = &DD7[1*numberOfLBnodes];
      D7.f[1] = &DD7[2*numberOfLBnodes];
      D7.f[4] = &DD7[3*numberOfLBnodes];
      D7.f[3] = &DD7[4*numberOfLBnodes];
      D7.f[6] = &DD7[5*numberOfLBnodes];
      D7.f[5] = &DD7[6*numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfLBnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D7.f[1])[ke   ]+ (D7.f[2])[kw  ]+ 
                        (D7.f[3])[kn   ]+ (D7.f[4])[ks  ]+
                        (D7.f[5])[kt   ]+ (D7.f[6])[kb  ]+
                        (D7.f[0])[kzero];  
      }
   }
}

























// DEPRECATED (2022)
//////////////////////////////////////////////////////////////////////////////////
// __global__ void LBCalcMacThS27(real* Conc,
//                                           unsigned int* geoD,
//                                           unsigned int* neighborX,
//                                           unsigned int* neighborY,
//                                           unsigned int* neighborZ,
//                                           unsigned int size_Mat,
//                                           real* DD27,
//                                           bool isEvenTimestep)
// {
//    Distributions27 D27;
//    if (isEvenTimestep==true)
//    {
//       D27.f[dP00] = &DD27[dP00 * size_Mat];
//       D27.f[dM00] = &DD27[dM00 * size_Mat];
//       D27.f[d0P0] = &DD27[d0P0 * size_Mat];
//       D27.f[d0M0] = &DD27[d0M0 * size_Mat];
//       D27.f[d00P] = &DD27[d00P * size_Mat];
//       D27.f[d00M] = &DD27[d00M * size_Mat];
//       D27.f[dPP0] = &DD27[dPP0 * size_Mat];
//       D27.f[dMM0] = &DD27[dMM0 * size_Mat];
//       D27.f[dPM0] = &DD27[dPM0 * size_Mat];
//       D27.f[dMP0] = &DD27[dMP0 * size_Mat];
//       D27.f[dP0P] = &DD27[dP0P * size_Mat];
//       D27.f[dM0M] = &DD27[dM0M * size_Mat];
//       D27.f[dP0M] = &DD27[dP0M * size_Mat];
//       D27.f[dM0P] = &DD27[dM0P * size_Mat];
//       D27.f[d0PP] = &DD27[d0PP * size_Mat];
//       D27.f[d0MM] = &DD27[d0MM * size_Mat];
//       D27.f[d0PM] = &DD27[d0PM * size_Mat];
//       D27.f[d0MP] = &DD27[d0MP * size_Mat];
//       D27.f[d000] = &DD27[d000 * size_Mat];
//       D27.f[dPPP] = &DD27[dPPP * size_Mat];
//       D27.f[dMMP] = &DD27[dMMP * size_Mat];
//       D27.f[dPMP] = &DD27[dPMP * size_Mat];
//       D27.f[dMPP] = &DD27[dMPP * size_Mat];
//       D27.f[dPPM] = &DD27[dPPM * size_Mat];
//       D27.f[dMMM] = &DD27[dMMM * size_Mat];
//       D27.f[dPMM] = &DD27[dPMM * size_Mat];
//       D27.f[dMPM] = &DD27[dMPM * size_Mat];
//    }
//    else
//    {
//       D27.f[dM00] = &DD27[dP00 * size_Mat];
//       D27.f[dP00] = &DD27[dM00 * size_Mat];
//       D27.f[d0M0] = &DD27[d0P0 * size_Mat];
//       D27.f[d0P0] = &DD27[d0M0 * size_Mat];
//       D27.f[d00M] = &DD27[d00P * size_Mat];
//       D27.f[d00P] = &DD27[d00M * size_Mat];
//       D27.f[dMM0] = &DD27[dPP0 * size_Mat];
//       D27.f[dPP0] = &DD27[dMM0 * size_Mat];
//       D27.f[dMP0] = &DD27[dPM0 * size_Mat];
//       D27.f[dPM0] = &DD27[dMP0 * size_Mat];
//       D27.f[dM0M] = &DD27[dP0P * size_Mat];
//       D27.f[dP0P] = &DD27[dM0M * size_Mat];
//       D27.f[dM0P] = &DD27[dP0M * size_Mat];
//       D27.f[dP0M] = &DD27[dM0P * size_Mat];
//       D27.f[d0MM] = &DD27[d0PP * size_Mat];
//       D27.f[d0PP] = &DD27[d0MM * size_Mat];
//       D27.f[d0MP] = &DD27[d0PM * size_Mat];
//       D27.f[d0PM] = &DD27[d0MP * size_Mat];
//       D27.f[d000] = &DD27[d000 * size_Mat];
//       D27.f[dMMM] = &DD27[dPPP * size_Mat];
//       D27.f[dPPM] = &DD27[dMMP * size_Mat];
//       D27.f[dMPM] = &DD27[dPMP * size_Mat];
//       D27.f[dPMM] = &DD27[dMPP * size_Mat];
//       D27.f[dMMP] = &DD27[dPPM * size_Mat];
//       D27.f[dPPP] = &DD27[dMMM * size_Mat];
//       D27.f[dMPP] = &DD27[dPMM * size_Mat];
//       D27.f[dPMP] = &DD27[dMPM * size_Mat];
//    }
//    ////////////////////////////////////////////////////////////////////////////////
//    const unsigned  x = threadIdx.x;  // Globaler x-Index 
//    const unsigned  y = blockIdx.x;   // Globaler y-Index 
//    const unsigned  z = blockIdx.y;   // Globaler z-Index 

//    const unsigned nx = blockDim.x;
//    const unsigned ny = gridDim.x;

//    const unsigned k = nx*(ny*z + y) + x;
//    //////////////////////////////////////////////////////////////////////////

//    if(k<size_Mat)
//    {
//       //////////////////////////////////////////////////////////////////////////
//       //index
//       unsigned int kzero= k;
//       unsigned int ke   = k;
//       unsigned int kw   = neighborX[k];
//       unsigned int kn   = k;
//       unsigned int ks   = neighborY[k];
//       unsigned int kt   = k;
//       unsigned int kb   = neighborZ[k];
//       unsigned int ksw  = neighborY[kw];
//       unsigned int kne  = k;
//       unsigned int kse  = ks;
//       unsigned int knw  = kw;
//       unsigned int kbw  = neighborZ[kw];
//       unsigned int kte  = k;
//       unsigned int kbe  = kb;
//       unsigned int ktw  = kw;
//       unsigned int kbs  = neighborZ[ks];
//       unsigned int ktn  = k;
//       unsigned int kbn  = kb;
//       unsigned int kts  = ks;
//       unsigned int ktse = ks;
//       unsigned int kbnw = kbw;
//       unsigned int ktnw = kw;
//       unsigned int kbse = kbs;
//       unsigned int ktsw = ksw;
//       unsigned int kbne = kb;
//       unsigned int ktne = k;
//       unsigned int kbsw = neighborZ[ksw];
//       //////////////////////////////////////////////////////////////////////////
//       Conc[k] = c0o1;

//       if(geoD[k] == GEO_FLUID)
//       {
//          Conc[k]    =   (D27.f[dP00])[ke  ]+ (D27.f[dM00])[kw  ]+ 
//                         (D27.f[d0P0])[kn  ]+ (D27.f[d0M0])[ks  ]+
//                         (D27.f[d00P])[kt  ]+ (D27.f[d00M])[kb  ]+
//                         (D27.f[dPP0])[kne ]+ (D27.f[dMM0])[ksw ]+
//                         (D27.f[dPM0])[kse ]+ (D27.f[dMP0])[knw ]+
//                         (D27.f[dP0P])[kte ]+ (D27.f[dM0M])[kbw ]+
//                         (D27.f[dP0M])[kbe ]+ (D27.f[dM0P])[ktw ]+
//                         (D27.f[d0PP])[ktn ]+ (D27.f[d0MM])[kbs ]+
//                         (D27.f[d0PM])[kbn ]+ (D27.f[d0MP])[kts ]+
//                         (D27.f[d000])[kzero]+ 
//                         (D27.f[dPPP])[ktne]+ (D27.f[dMMP])[ktsw]+
//                         (D27.f[dPMP])[ktse]+ (D27.f[dMPP])[ktnw]+
//                         (D27.f[dPPM])[kbne]+ (D27.f[dMMM])[kbsw]+
//                         (D27.f[dPMM])[kbse]+ (D27.f[dMPM])[kbnw];
//       }
//    }   
// }



















////////////////////////////////////////////////////////////////////////////////
__global__ void GetPlaneConc7(real* Conc,
                                            int* kPC,
                                            unsigned int numberOfPointskPC,
                                            unsigned int* geoD,
                                            unsigned int* neighborX,
                                            unsigned int* neighborY,
                                            unsigned int* neighborZ,
                                            unsigned long long numberOfLBnodes,
                                            real* DD7,
                                            bool isEvenTimestep)
{
   Distributions7 D7;
   if (isEvenTimestep==true)
   {
      D7.f[0] = &DD7[0*numberOfLBnodes];
      D7.f[1] = &DD7[1*numberOfLBnodes];
      D7.f[2] = &DD7[2*numberOfLBnodes];
      D7.f[3] = &DD7[3*numberOfLBnodes];
      D7.f[4] = &DD7[4*numberOfLBnodes];
      D7.f[5] = &DD7[5*numberOfLBnodes];
      D7.f[6] = &DD7[6*numberOfLBnodes];
   } 
   else
   {
      D7.f[0] = &DD7[0*numberOfLBnodes];
      D7.f[2] = &DD7[1*numberOfLBnodes];
      D7.f[1] = &DD7[2*numberOfLBnodes];
      D7.f[4] = &DD7[3*numberOfLBnodes];
      D7.f[3] = &DD7[4*numberOfLBnodes];
      D7.f[6] = &DD7[5*numberOfLBnodes];
      D7.f[5] = &DD7[6*numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfPointskPC)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kPC[k];
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D7.f[1])[ke   ]+ (D7.f[2])[kw  ]+ 
                        (D7.f[3])[kn   ]+ (D7.f[4])[ks  ]+
                        (D7.f[5])[kt   ]+ (D7.f[6])[kb  ]+
                        (D7.f[0])[kzero];  
      }
   }
}






































////////////////////////////////////////////////////////////////////////////////
__global__ void GetPlaneConc27(real* Conc,
                                             int* kPC,
                                             unsigned int numberOfPointskPC,
                                             unsigned int* geoD,
                                             unsigned int* neighborX,
                                             unsigned int* neighborY,
                                             unsigned int* neighborZ,
                                             unsigned long long numberOfLBnodes,
                                             real* DD27,
                                             bool isEvenTimestep)
{
   Distributions27 D27;
   if (isEvenTimestep==true)
   {
      D27.f[dP00] = &DD27[dP00 * numberOfLBnodes];
      D27.f[dM00] = &DD27[dM00 * numberOfLBnodes];
      D27.f[d0P0] = &DD27[d0P0 * numberOfLBnodes];
      D27.f[d0M0] = &DD27[d0M0 * numberOfLBnodes];
      D27.f[d00P] = &DD27[d00P * numberOfLBnodes];
      D27.f[d00M] = &DD27[d00M * numberOfLBnodes];
      D27.f[dPP0] = &DD27[dPP0 * numberOfLBnodes];
      D27.f[dMM0] = &DD27[dMM0 * numberOfLBnodes];
      D27.f[dPM0] = &DD27[dPM0 * numberOfLBnodes];
      D27.f[dMP0] = &DD27[dMP0 * numberOfLBnodes];
      D27.f[dP0P] = &DD27[dP0P * numberOfLBnodes];
      D27.f[dM0M] = &DD27[dM0M * numberOfLBnodes];
      D27.f[dP0M] = &DD27[dP0M * numberOfLBnodes];
      D27.f[dM0P] = &DD27[dM0P * numberOfLBnodes];
      D27.f[d0PP] = &DD27[d0PP * numberOfLBnodes];
      D27.f[d0MM] = &DD27[d0MM * numberOfLBnodes];
      D27.f[d0PM] = &DD27[d0PM * numberOfLBnodes];
      D27.f[d0MP] = &DD27[d0MP * numberOfLBnodes];
      D27.f[d000] = &DD27[d000 * numberOfLBnodes];
      D27.f[dPPP] = &DD27[dPPP * numberOfLBnodes];
      D27.f[dMMP] = &DD27[dMMP * numberOfLBnodes];
      D27.f[dPMP] = &DD27[dPMP * numberOfLBnodes];
      D27.f[dMPP] = &DD27[dMPP * numberOfLBnodes];
      D27.f[dPPM] = &DD27[dPPM * numberOfLBnodes];
      D27.f[dMMM] = &DD27[dMMM * numberOfLBnodes];
      D27.f[dPMM] = &DD27[dPMM * numberOfLBnodes];
      D27.f[dMPM] = &DD27[dMPM * numberOfLBnodes];
   }
   else
   {
      D27.f[dM00] = &DD27[dP00 * numberOfLBnodes];
      D27.f[dP00] = &DD27[dM00 * numberOfLBnodes];
      D27.f[d0M0] = &DD27[d0P0 * numberOfLBnodes];
      D27.f[d0P0] = &DD27[d0M0 * numberOfLBnodes];
      D27.f[d00M] = &DD27[d00P * numberOfLBnodes];
      D27.f[d00P] = &DD27[d00M * numberOfLBnodes];
      D27.f[dMM0] = &DD27[dPP0 * numberOfLBnodes];
      D27.f[dPP0] = &DD27[dMM0 * numberOfLBnodes];
      D27.f[dMP0] = &DD27[dPM0 * numberOfLBnodes];
      D27.f[dPM0] = &DD27[dMP0 * numberOfLBnodes];
      D27.f[dM0M] = &DD27[dP0P * numberOfLBnodes];
      D27.f[dP0P] = &DD27[dM0M * numberOfLBnodes];
      D27.f[dM0P] = &DD27[dP0M * numberOfLBnodes];
      D27.f[dP0M] = &DD27[dM0P * numberOfLBnodes];
      D27.f[d0MM] = &DD27[d0PP * numberOfLBnodes];
      D27.f[d0PP] = &DD27[d0MM * numberOfLBnodes];
      D27.f[d0MP] = &DD27[d0PM * numberOfLBnodes];
      D27.f[d0PM] = &DD27[d0MP * numberOfLBnodes];
      D27.f[d000] = &DD27[d000 * numberOfLBnodes];
      D27.f[dMMM] = &DD27[dPPP * numberOfLBnodes];
      D27.f[dPPM] = &DD27[dMMP * numberOfLBnodes];
      D27.f[dMPM] = &DD27[dPMP * numberOfLBnodes];
      D27.f[dPMM] = &DD27[dMPP * numberOfLBnodes];
      D27.f[dMMP] = &DD27[dPPM * numberOfLBnodes];
      D27.f[dPPP] = &DD27[dMMM * numberOfLBnodes];
      D27.f[dMPP] = &DD27[dPMM * numberOfLBnodes];
      D27.f[dPMP] = &DD27[dMPM * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfPointskPC)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kPC[k];
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = kzero;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = kzero;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = kzero;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = kzero;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = c0o1;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D27.f[dP00])[ke  ]+ (D27.f[dM00])[kw  ]+ 
                        (D27.f[d0P0])[kn  ]+ (D27.f[d0M0])[ks  ]+
                        (D27.f[d00P])[kt  ]+ (D27.f[d00M])[kb  ]+
                        (D27.f[dPP0])[kne ]+ (D27.f[dMM0])[ksw ]+
                        (D27.f[dPM0])[kse ]+ (D27.f[dMP0])[knw ]+
                        (D27.f[dP0P])[kte ]+ (D27.f[dM0M])[kbw ]+
                        (D27.f[dP0M])[kbe ]+ (D27.f[dM0P])[ktw ]+
                        (D27.f[d0PP])[ktn ]+ (D27.f[d0MM])[kbs ]+
                        (D27.f[d0PM])[kbn ]+ (D27.f[d0MP])[kts ]+
                        (D27.f[d000])[kzero]+ 
                        (D27.f[dPPP])[ktne]+ (D27.f[dMMP])[ktsw]+
                        (D27.f[dPMP])[ktse]+ (D27.f[dMPP])[ktnw]+
                        (D27.f[dPPM])[kbne]+ (D27.f[dMMM])[kbsw]+
                        (D27.f[dPMM])[kbse]+ (D27.f[dMPM])[kbnw];
      }
   }   
}

void PlaneConcThS7(real* Conc, int* kPC, unsigned int numberOfPointskPC, unsigned int* geoD, unsigned int* neighborX,
                   unsigned int* neighborY, unsigned int* neighborZ, unsigned long long numberOfLBnodes,
                   unsigned int numberOfThreads, real* DD7, bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfPointskPC);

   GetPlaneConc7<<<grid.grid, grid.threads>>>(Conc, kPC, numberOfPointskPC, geoD, neighborX, neighborY, neighborZ,
                                              numberOfLBnodes, DD7, isEvenTimestep);
   getLastCudaError("GetPlaneConc7 execution failed");
}
//////////////////////////////////////////////////////////////////////////
void PlaneConcThS27(real* Conc, int* kPC, unsigned int numberOfPointskPC, unsigned int* geoD, unsigned int* neighborX,
                    unsigned int* neighborY, unsigned int* neighborZ, unsigned long long numberOfLBnodes,
                    unsigned int numberOfThreads, real* DD27, bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfPointskPC);

   GetPlaneConc27<<<grid.grid, grid.threads>>>(Conc, kPC, numberOfPointskPC, geoD, neighborX, neighborY, neighborZ,
                                               numberOfLBnodes, DD27, isEvenTimestep);
   getLastCudaError("GetPlaneConc27 execution failed");
}

void CalcConcentration27(unsigned int numberOfThreads, real* Conc, unsigned int* geoD, unsigned int* neighborX,
                         unsigned int* neighborY, unsigned int* neighborZ, unsigned long long numberOfLBnodes, real* DD27,
                         bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfLBnodes);

   CalcConc27<<<grid.grid, grid.threads>>>(Conc, geoD, neighborX, neighborY, neighborZ, numberOfLBnodes, DD27,
                                           isEvenTimestep);
   getLastCudaError("CalcConc27 execution failed");
}

void CalcConcThS7(real* Conc, unsigned int* geoD, unsigned int* neighborX, unsigned int* neighborY, unsigned int* neighborZ,
                  unsigned long long numberOfLBnodes, unsigned int numberOfThreads, real* DD7, bool isEvenTimestep)
{
   vf::cuda::CudaGrid grid = vf::cuda::CudaGrid(numberOfThreads, numberOfLBnodes);

   CalcConc7<<<grid.grid, grid.threads>>>(Conc, geoD, neighborX, neighborY, neighborZ, numberOfLBnodes, DD7, isEvenTimestep);
   getLastCudaError("CalcConc7 execution failed");
}
