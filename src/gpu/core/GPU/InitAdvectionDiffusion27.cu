#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file InitAdvectionDiffusion.cu
//! \ingroup GPU
//! \author Martin Schoenherr
//=======================================================================================
/* Device code */
#include "LBM/LB.h"
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;

__global__ void InitAD27(
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	uint* typeOfGridNode,
	real* concentration,
	real* velocityX,
	real* velocityY,
	real* velocityZ,
	unsigned long long numberOfLBnodes,
	real* distributionsAD,
	bool isEvenTimestep)
{
	//////////////////////////////////////////////////////////////////////////
	//! The initialization is executed in the following steps
	//!
	////////////////////////////////////////////////////////////////////////////////
	//! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
	//!
	const unsigned  x = threadIdx.x;  // Globaler x-Index
	const unsigned  y = blockIdx.x;   // Globaler y-Index
	const unsigned  z = blockIdx.y;   // Globaler z-Index

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;

	//////////////////////////////////////////////////////////////////////////
	// run for all indices in size_Mat and fluid nodes
	if ((k < numberOfLBnodes) && (typeOfGridNode[k] == GEO_FLUID))
	{
		//////////////////////////////////////////////////////////////////////////
		//! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
		//! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
		//!
		Distributions27 distAD;
		if (isEvenTimestep)
		{
			distAD.f[dP00] = &distributionsAD[dP00 * numberOfLBnodes];
			distAD.f[dM00] = &distributionsAD[dM00 * numberOfLBnodes];
			distAD.f[d0P0] = &distributionsAD[d0P0 * numberOfLBnodes];
			distAD.f[d0M0] = &distributionsAD[d0M0 * numberOfLBnodes];
			distAD.f[d00P] = &distributionsAD[d00P * numberOfLBnodes];
			distAD.f[d00M] = &distributionsAD[d00M * numberOfLBnodes];
			distAD.f[dPP0] = &distributionsAD[dPP0 * numberOfLBnodes];
			distAD.f[dMM0] = &distributionsAD[dMM0 * numberOfLBnodes];
			distAD.f[dPM0] = &distributionsAD[dPM0 * numberOfLBnodes];
			distAD.f[dMP0] = &distributionsAD[dMP0 * numberOfLBnodes];
			distAD.f[dP0P] = &distributionsAD[dP0P * numberOfLBnodes];
			distAD.f[dM0M] = &distributionsAD[dM0M * numberOfLBnodes];
			distAD.f[dP0M] = &distributionsAD[dP0M * numberOfLBnodes];
			distAD.f[dM0P] = &distributionsAD[dM0P * numberOfLBnodes];
			distAD.f[d0PP] = &distributionsAD[d0PP * numberOfLBnodes];
			distAD.f[d0MM] = &distributionsAD[d0MM * numberOfLBnodes];
			distAD.f[d0PM] = &distributionsAD[d0PM * numberOfLBnodes];
			distAD.f[d0MP] = &distributionsAD[d0MP * numberOfLBnodes];
			distAD.f[d000] = &distributionsAD[d000 * numberOfLBnodes];
			distAD.f[dPPP] = &distributionsAD[dPPP * numberOfLBnodes];
			distAD.f[dMMP] = &distributionsAD[dMMP * numberOfLBnodes];
			distAD.f[dPMP] = &distributionsAD[dPMP * numberOfLBnodes];
			distAD.f[dMPP] = &distributionsAD[dMPP * numberOfLBnodes];
			distAD.f[dPPM] = &distributionsAD[dPPM * numberOfLBnodes];
			distAD.f[dMMM] = &distributionsAD[dMMM * numberOfLBnodes];
			distAD.f[dPMM] = &distributionsAD[dPMM * numberOfLBnodes];
			distAD.f[dMPM] = &distributionsAD[dMPM * numberOfLBnodes];
		}
		else
		{
			distAD.f[dM00] = &distributionsAD[dP00 * numberOfLBnodes];
			distAD.f[dP00] = &distributionsAD[dM00 * numberOfLBnodes];
			distAD.f[d0M0] = &distributionsAD[d0P0 * numberOfLBnodes];
			distAD.f[d0P0] = &distributionsAD[d0M0 * numberOfLBnodes];
			distAD.f[d00M] = &distributionsAD[d00P * numberOfLBnodes];
			distAD.f[d00P] = &distributionsAD[d00M * numberOfLBnodes];
			distAD.f[dMM0] = &distributionsAD[dPP0 * numberOfLBnodes];
			distAD.f[dPP0] = &distributionsAD[dMM0 * numberOfLBnodes];
			distAD.f[dMP0] = &distributionsAD[dPM0 * numberOfLBnodes];
			distAD.f[dPM0] = &distributionsAD[dMP0 * numberOfLBnodes];
			distAD.f[dM0M] = &distributionsAD[dP0P * numberOfLBnodes];
			distAD.f[dP0P] = &distributionsAD[dM0M * numberOfLBnodes];
			distAD.f[dM0P] = &distributionsAD[dP0M * numberOfLBnodes];
			distAD.f[dP0M] = &distributionsAD[dM0P * numberOfLBnodes];
			distAD.f[d0MM] = &distributionsAD[d0PP * numberOfLBnodes];
			distAD.f[d0PP] = &distributionsAD[d0MM * numberOfLBnodes];
			distAD.f[d0MP] = &distributionsAD[d0PM * numberOfLBnodes];
			distAD.f[d0PM] = &distributionsAD[d0MP * numberOfLBnodes];
			distAD.f[d000] = &distributionsAD[d000 * numberOfLBnodes];
			distAD.f[dMMM] = &distributionsAD[dPPP * numberOfLBnodes];
			distAD.f[dPPM] = &distributionsAD[dMMP * numberOfLBnodes];
			distAD.f[dMPM] = &distributionsAD[dPMP * numberOfLBnodes];
			distAD.f[dPMM] = &distributionsAD[dMPP * numberOfLBnodes];
			distAD.f[dMMP] = &distributionsAD[dPPM * numberOfLBnodes];
			distAD.f[dPPP] = &distributionsAD[dMMM * numberOfLBnodes];
			distAD.f[dMPP] = &distributionsAD[dPMM * numberOfLBnodes];
			distAD.f[dPMP] = &distributionsAD[dMPM * numberOfLBnodes];
		}
		//////////////////////////////////////////////////////////////////////////
		//! - Set local velocities and concetration
		//!
		real conc = concentration[k];
		real  vx1 = velocityX[k];
		real  vx2 = velocityY[k];
		real  vx3 = velocityZ[k];
		//////////////////////////////////////////////////////////////////////////
		//! - Set neighbor indices (necessary for indirect addressing)
		//!
		uint kzero = k;
		uint ke    = k;
		uint kw    = neighborX[k];
		uint kn    = k;
		uint ks    = neighborY[k];
		uint kt    = k;
		uint kb    = neighborZ[k];
		uint ksw   = neighborY[kw];
		uint kne   = k;
		uint kse   = ks;
		uint knw   = kw;
		uint kbw   = neighborZ[kw];
		uint kte   = k;
		uint kbe   = kb;
		uint ktw   = kw;
		uint kbs   = neighborZ[ks];
		uint ktn   = k;
		uint kbn   = kb;
		uint kts   = ks;
		uint ktse  = ks;
		uint kbnw  = kbw;
		uint ktnw  = kw;
		uint kbse  = kbs;
		uint ktsw  = ksw;
		uint kbne  = kb;
		uint ktne  = k;
		uint kbsw  = neighborZ[ksw];
		//////////////////////////////////////////////////////////////////////////
		//! - Calculate the equilibrium and set the distributions
		//!
		real cu_sq = c3o2*(vx1*vx1 + vx2*vx2 + vx3*vx3);

		(distAD.f[d000])[kzero] = c8o27  * conc * (c1o1 - cu_sq);
		(distAD.f[dP00])[ke   ] = c2o27  * conc * (c1o1 + c3o1 * ( vx1            ) + c9o2 * ( vx1            ) * ( vx1            ) - cu_sq);
		(distAD.f[dM00])[kw   ] = c2o27  * conc * (c1o1 + c3o1 * (-vx1            ) + c9o2 * (-vx1            ) * (-vx1            ) - cu_sq);
		(distAD.f[d0P0])[kn   ] = c2o27  * conc * (c1o1 + c3o1 * (       vx2      ) + c9o2 * (       vx2      ) * (       vx2      ) - cu_sq);
		(distAD.f[d0M0])[ks   ] = c2o27  * conc * (c1o1 + c3o1 * (     - vx2      ) + c9o2 * (     - vx2      ) * (     - vx2      ) - cu_sq);
		(distAD.f[d00P])[kt   ] = c2o27  * conc * (c1o1 + c3o1 * (             vx3) + c9o2 * (             vx3) * (             vx3) - cu_sq);
		(distAD.f[d00M])[kb   ] = c2o27  * conc * (c1o1 + c3o1 * (           - vx3) + c9o2 * (           - vx3) * (           - vx3) - cu_sq);
		(distAD.f[dPP0])[kne  ] = c1o54  * conc * (c1o1 + c3o1 * ( vx1 + vx2      ) + c9o2 * ( vx1 + vx2      ) * ( vx1 + vx2      ) - cu_sq);
		(distAD.f[dMM0])[ksw  ] = c1o54  * conc * (c1o1 + c3o1 * (-vx1 - vx2      ) + c9o2 * (-vx1 - vx2      ) * (-vx1 - vx2      ) - cu_sq);
		(distAD.f[dPM0])[kse  ] = c1o54  * conc * (c1o1 + c3o1 * ( vx1 - vx2      ) + c9o2 * ( vx1 - vx2      ) * ( vx1 - vx2      ) - cu_sq);
		(distAD.f[dMP0])[knw  ] = c1o54  * conc * (c1o1 + c3o1 * (-vx1 + vx2      ) + c9o2 * (-vx1 + vx2      ) * (-vx1 + vx2      ) - cu_sq);
		(distAD.f[dP0P])[kte  ] = c1o54  * conc * (c1o1 + c3o1 * ( vx1       + vx3) + c9o2 * ( vx1       + vx3) * ( vx1       + vx3) - cu_sq);
		(distAD.f[dM0M])[kbw  ] = c1o54  * conc * (c1o1 + c3o1 * (-vx1       - vx3) + c9o2 * (-vx1       - vx3) * (-vx1       - vx3) - cu_sq);
		(distAD.f[dP0M])[kbe  ] = c1o54  * conc * (c1o1 + c3o1 * ( vx1       - vx3) + c9o2 * ( vx1       - vx3) * ( vx1       - vx3) - cu_sq);
		(distAD.f[dM0P])[ktw  ] = c1o54  * conc * (c1o1 + c3o1 * (-vx1       + vx3) + c9o2 * (-vx1       + vx3) * (-vx1       + vx3) - cu_sq);
		(distAD.f[d0PP])[ktn  ] = c1o54  * conc * (c1o1 + c3o1 * (       vx2 + vx3) + c9o2 * (       vx2 + vx3) * (       vx2 + vx3) - cu_sq);
		(distAD.f[d0MM])[kbs  ] = c1o54  * conc * (c1o1 + c3o1 * (     - vx2 - vx3) + c9o2 * (     - vx2 - vx3) * (     - vx2 - vx3) - cu_sq);
		(distAD.f[d0PM])[kbn  ] = c1o54  * conc * (c1o1 + c3o1 * (       vx2 - vx3) + c9o2 * (       vx2 - vx3) * (       vx2 - vx3) - cu_sq);
		(distAD.f[d0MP])[kts  ] = c1o54  * conc * (c1o1 + c3o1 * (     - vx2 + vx3) + c9o2 * (     - vx2 + vx3) * (     - vx2 + vx3) - cu_sq);
		(distAD.f[dPPP])[ktne ] = c1o216 * conc * (c1o1 + c3o1 * ( vx1 + vx2 + vx3) + c9o2 * ( vx1 + vx2 + vx3) * ( vx1 + vx2 + vx3) - cu_sq);
		(distAD.f[dMMM])[kbsw ] = c1o216 * conc * (c1o1 + c3o1 * (-vx1 - vx2 - vx3) + c9o2 * (-vx1 - vx2 - vx3) * (-vx1 - vx2 - vx3) - cu_sq);
		(distAD.f[dPPM])[kbne ] = c1o216 * conc * (c1o1 + c3o1 * ( vx1 + vx2 - vx3) + c9o2 * ( vx1 + vx2 - vx3) * ( vx1 + vx2 - vx3) - cu_sq);
		(distAD.f[dMMP])[ktsw ] = c1o216 * conc * (c1o1 + c3o1 * (-vx1 - vx2 + vx3) + c9o2 * (-vx1 - vx2 + vx3) * (-vx1 - vx2 + vx3) - cu_sq);
		(distAD.f[dPMP])[ktse ] = c1o216 * conc * (c1o1 + c3o1 * ( vx1 - vx2 + vx3) + c9o2 * ( vx1 - vx2 + vx3) * ( vx1 - vx2 + vx3) - cu_sq);
		(distAD.f[dMPM])[kbnw ] = c1o216 * conc * (c1o1 + c3o1 * (-vx1 + vx2 - vx3) + c9o2 * (-vx1 + vx2 - vx3) * (-vx1 + vx2 - vx3) - cu_sq);
		(distAD.f[dPMM])[kbse ] = c1o216 * conc * (c1o1 + c3o1 * ( vx1 - vx2 - vx3) + c9o2 * ( vx1 - vx2 - vx3) * ( vx1 - vx2 - vx3) - cu_sq);
		(distAD.f[dMPP])[ktnw ] = c1o216 * conc * (c1o1 + c3o1 * (-vx1 + vx2 + vx3) + c9o2 * (-vx1 + vx2 + vx3) * (-vx1 + vx2 + vx3) - cu_sq);
	}
}




















// DEPRECATED (2022)

// ////////////////////////////////////////////////////////////////////////////////
// __global__ void InitAD27(unsigned int* neighborX,
//                                        unsigned int* neighborY,
//                                        unsigned int* neighborZ,
//                                        unsigned int* geoD,
//                                        real* Conc,
//                                        real* ux,
//                                        real* uy,
//                                        real* uz,
//                                        unsigned int size_Mat,
//                                        real* DD27,
//                                        bool EvenOrOdd)
// {
//    ////////////////////////////////////////////////////////////////////////////////
//    const unsigned  x = threadIdx.x;  // Globaler x-Index 
//    const unsigned  y = blockIdx.x;   // Globaler y-Index 
//    const unsigned  z = blockIdx.y;   // Globaler z-Index 

//    const unsigned nx = blockDim.x;
//    const unsigned ny = gridDim.x;

//    const unsigned k = nx*(ny*z + y) + x;
//    //////////////////////////////////////////////////////////////////////////

//    if(k<size_Mat)
//    {
//       ////////////////////////////////////////////////////////////////////////////////
//       unsigned int BC;
//       BC        =   geoD[k];

//       if( BC != GEO_SOLID && BC != GEO_VOID)
//       {
//          Distributions27 D27;
//          if (EvenOrOdd==true)
//          {
//             D27.f[dP00] = &DD27[dP00 * size_Mat];
//             D27.f[dM00] = &DD27[dM00 * size_Mat];
//             D27.f[d0P0] = &DD27[d0P0 * size_Mat];
//             D27.f[d0M0] = &DD27[d0M0 * size_Mat];
//             D27.f[d00P] = &DD27[d00P * size_Mat];
//             D27.f[d00M] = &DD27[d00M * size_Mat];
//             D27.f[dPP0] = &DD27[dPP0 * size_Mat];
//             D27.f[dMM0] = &DD27[dMM0 * size_Mat];
//             D27.f[dPM0] = &DD27[dPM0 * size_Mat];
//             D27.f[dMP0] = &DD27[dMP0 * size_Mat];
//             D27.f[dP0P] = &DD27[dP0P * size_Mat];
//             D27.f[dM0M] = &DD27[dM0M * size_Mat];
//             D27.f[dP0M] = &DD27[dP0M * size_Mat];
//             D27.f[dM0P] = &DD27[dM0P * size_Mat];
//             D27.f[d0PP] = &DD27[d0PP * size_Mat];
//             D27.f[d0MM] = &DD27[d0MM * size_Mat];
//             D27.f[d0PM] = &DD27[d0PM * size_Mat];
//             D27.f[d0MP] = &DD27[d0MP * size_Mat];
//             D27.f[d000] = &DD27[d000 * size_Mat];
//             D27.f[dPPP] = &DD27[dPPP * size_Mat];
//             D27.f[dMMP] = &DD27[dMMP * size_Mat];
//             D27.f[dPMP] = &DD27[dPMP * size_Mat];
//             D27.f[dMPP] = &DD27[dMPP * size_Mat];
//             D27.f[dPPM] = &DD27[dPPM * size_Mat];
//             D27.f[dMMM] = &DD27[dMMM * size_Mat];
//             D27.f[dPMM] = &DD27[dPMM * size_Mat];
//             D27.f[dMPM] = &DD27[dMPM * size_Mat];
//          }
//          else
//          {
//             D27.f[dM00] = &DD27[dP00 * size_Mat];
//             D27.f[dP00] = &DD27[dM00 * size_Mat];
//             D27.f[d0M0] = &DD27[d0P0 * size_Mat];
//             D27.f[d0P0] = &DD27[d0M0 * size_Mat];
//             D27.f[d00M] = &DD27[d00P * size_Mat];
//             D27.f[d00P] = &DD27[d00M * size_Mat];
//             D27.f[dMM0] = &DD27[dPP0 * size_Mat];
//             D27.f[dPP0] = &DD27[dMM0 * size_Mat];
//             D27.f[dMP0] = &DD27[dPM0 * size_Mat];
//             D27.f[dPM0] = &DD27[dMP0 * size_Mat];
//             D27.f[dM0M] = &DD27[dP0P * size_Mat];
//             D27.f[dP0P] = &DD27[dM0M * size_Mat];
//             D27.f[dM0P] = &DD27[dP0M * size_Mat];
//             D27.f[dP0M] = &DD27[dM0P * size_Mat];
//             D27.f[d0MM] = &DD27[d0PP * size_Mat];
//             D27.f[d0PP] = &DD27[d0MM * size_Mat];
//             D27.f[d0MP] = &DD27[d0PM * size_Mat];
//             D27.f[d0PM] = &DD27[d0MP * size_Mat];
//             D27.f[d000] = &DD27[d000 * size_Mat];
//             D27.f[dMMM] = &DD27[dPPP * size_Mat];
//             D27.f[dPPM] = &DD27[dMMP * size_Mat];
//             D27.f[dMPM] = &DD27[dPMP * size_Mat];
//             D27.f[dPMM] = &DD27[dMPP * size_Mat];
//             D27.f[dMMP] = &DD27[dPPM * size_Mat];
//             D27.f[dPPP] = &DD27[dMMM * size_Mat];
//             D27.f[dMPP] = &DD27[dPMM * size_Mat];
//             D27.f[dPMP] = &DD27[dMPM * size_Mat];
//          }
//          //////////////////////////////////////////////////////////////////////////
//          real ConcD = Conc[k];
//          real   vx1 = ux[k];
//          real   vx2 = uy[k];
//          real   vx3 = uz[k];
//          //real lambdaD     = -three + sqrt(three);
//          //real Diffusivity = c1o20;
//          //real Lam         = -(c1o2+one/lambdaD);
//          //real nue_d       = Lam/three;
//          //real ae          = Diffusivity/nue_d - one;
//          //real ux_sq       = vx1 * vx1;
//          //real uy_sq       = vx2 * vx2;
//          //real uz_sq       = vx3 * vx3;
//          ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          //D3Q7
//          ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          //index
//          //unsigned int kzero= k;
//          //unsigned int ke   = k;
//          //unsigned int kw   = neighborX[k];
//          //unsigned int kn   = k;
//          //unsigned int ks   = neighborY[k];
//          //unsigned int kt   = k;
//          //unsigned int kb   = neighborZ[k];
//          //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          //(D7.f[0])[kzero] = ConcD*(c1o3*(ae*(-three))-(ux_sq+uy_sq+uz_sq));
//          //(D7.f[1])[ke   ] = ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)+vx1*c1o2);
//          //(D7.f[2])[kw   ] = ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)-vx1*c1o2);
//          //(D7.f[3])[kn   ] = ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)+vx2*c1o2);
//          //(D7.f[4])[ks   ] = ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)-vx2*c1o2);
//          //(D7.f[5])[kt   ] = ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)+vx3*c1o2);
//          //(D7.f[6])[kb   ] = ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)-vx3*c1o2);
//          ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//          ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          //D3Q27
//          ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          //index
//          unsigned int kzero= k;
//          unsigned int ke   = k;
//          unsigned int kw   = neighborX[k];
//          unsigned int kn   = k;
//          unsigned int ks   = neighborY[k];
//          unsigned int kt   = k;
//          unsigned int kb   = neighborZ[k];
//          unsigned int ksw  = neighborY[kw];
//          unsigned int kne  = k;
//          unsigned int kse  = ks;
//          unsigned int knw  = kw;
//          unsigned int kbw  = neighborZ[kw];
//          unsigned int kte  = k;
//          unsigned int kbe  = kb;
//          unsigned int ktw  = kw;
//          unsigned int kbs  = neighborZ[ks];
//          unsigned int ktn  = k;
//          unsigned int kbn  = kb;
//          unsigned int kts  = ks;
//          unsigned int ktse = ks;
//          unsigned int kbnw = kbw;
//          unsigned int ktnw = kw;
//          unsigned int kbse = kbs;
//          unsigned int ktsw = ksw;
//          unsigned int kbne = kb;
//          unsigned int ktne = k;
//          unsigned int kbsw = neighborZ[ksw];
//          ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

//          (D27.f[d000])[kzero] =   c8o27* ConcD*(c1o1-cu_sq);
//          (D27.f[dP00])[ke   ] =   c2o27* ConcD*(c1o1+c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq);
//          (D27.f[dM00])[kw   ] =   c2o27* ConcD*(c1o1+c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq);
//          (D27.f[d0P0])[kn   ] =   c2o27* ConcD*(c1o1+c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq);
//          (D27.f[d0M0])[ks   ] =   c2o27* ConcD*(c1o1+c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq);
//          (D27.f[d00P])[kt   ] =   c2o27* ConcD*(c1o1+c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq);
//          (D27.f[d00M])[kb   ] =   c2o27* ConcD*(c1o1+c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq);
//          (D27.f[dPP0])[kne  ] =   c1o54* ConcD*(c1o1+c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq);
//          (D27.f[dMM0])[ksw  ] =   c1o54* ConcD*(c1o1+c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq);
//          (D27.f[dPM0])[kse  ] =   c1o54* ConcD*(c1o1+c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq);
//          (D27.f[dMP0])[knw  ] =   c1o54* ConcD*(c1o1+c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq);
//          (D27.f[dP0P])[kte  ] =   c1o54* ConcD*(c1o1+c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq);
//          (D27.f[dM0M])[kbw  ] =   c1o54* ConcD*(c1o1+c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq);
//          (D27.f[dP0M])[kbe  ] =   c1o54* ConcD*(c1o1+c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq);
//          (D27.f[dM0P])[ktw  ] =   c1o54* ConcD*(c1o1+c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq);
//          (D27.f[d0PP])[ktn  ] =   c1o54* ConcD*(c1o1+c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq);
//          (D27.f[d0MM])[kbs  ] =   c1o54* ConcD*(c1o1+c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq);
//          (D27.f[d0PM])[kbn  ] =   c1o54* ConcD*(c1o1+c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq);
//          (D27.f[d0MP])[kts  ] =   c1o54* ConcD*(c1o1+c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq);
//          (D27.f[dPPP])[ktne ] =   c1o216*ConcD*(c1o1+c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq);
//          (D27.f[dMMM])[kbsw ] =   c1o216*ConcD*(c1o1+c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq);
//          (D27.f[dPPM])[kbne ] =   c1o216*ConcD*(c1o1+c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq);
//          (D27.f[dMMP])[ktsw ] =   c1o216*ConcD*(c1o1+c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq);
//          (D27.f[dPMP])[ktse ] =   c1o216*ConcD*(c1o1+c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq);
//          (D27.f[dMPM])[kbnw ] =   c1o216*ConcD*(c1o1+c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq);
//          (D27.f[dPMM])[kbse ] =   c1o216*ConcD*(c1o1+c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq);
//          (D27.f[dMPP])[ktnw ] =   c1o216*ConcD*(c1o1+c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq);
//          ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//       }
//    }
// }


















////////////////////////////////////////////////////////////////////////////////
__global__ void InitAD7( unsigned int* neighborX,
                                    unsigned int* neighborY,
                                    unsigned int* neighborZ,
                                    unsigned int* geoD,
                                    real* Conc,
                                    real* ux,
                                    real* uy,
                                    real* uz,
                                    unsigned long long numberOfLBnodes,
                                    real* DD7,
                                    bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfLBnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   geoD[k];

      if( BC != GEO_SOLID && BC != GEO_VOID)
      {
         Distributions7 D7;
         if (EvenOrOdd==true)
         {
            D7.f[0] = &DD7[0*numberOfLBnodes];
            D7.f[1] = &DD7[1*numberOfLBnodes];
            D7.f[2] = &DD7[2*numberOfLBnodes];
            D7.f[3] = &DD7[3*numberOfLBnodes];
            D7.f[4] = &DD7[4*numberOfLBnodes];
            D7.f[5] = &DD7[5*numberOfLBnodes];
            D7.f[6] = &DD7[6*numberOfLBnodes];
         }
         else
         {
            D7.f[0] = &DD7[0*numberOfLBnodes];
            D7.f[2] = &DD7[1*numberOfLBnodes];
            D7.f[1] = &DD7[2*numberOfLBnodes];
            D7.f[4] = &DD7[3*numberOfLBnodes];
            D7.f[3] = &DD7[4*numberOfLBnodes];
            D7.f[6] = &DD7[5*numberOfLBnodes];
            D7.f[5] = &DD7[6*numberOfLBnodes];
         }
         //////////////////////////////////////////////////////////////////////////
         real ConcD = Conc[k];
         real   vx1 = ux[k];
         real   vx2 = uy[k];
         real   vx3 = uz[k];
         real lambdaD     = -c3o1 + sqrt(c3o1);
         real Diffusivity = c1o20;
         real Lam         = -(c1o2+c1o1/lambdaD);
         real nue_d       = Lam/c3o1;
         real ae          = Diffusivity/nue_d - c1o1;
         real ux_sq       = vx1 * vx1;
         real uy_sq       = vx2 * vx2;
         real uz_sq       = vx3 * vx3;
         //////////////////////////////////////////////////////////////////////////
         //index
         //////////////////////////////////////////////////////////////////////////
         unsigned int kzero= k;
         unsigned int ke   = k;
         unsigned int kw   = neighborX[k];
         unsigned int kn   = k;
         unsigned int ks   = neighborY[k];
         unsigned int kt   = k;
         unsigned int kb   = neighborZ[k];
         //////////////////////////////////////////////////////////////////////////

         (D7.f[0])[kzero] = ConcD*(c1o3*(ae*(-c3o1))-(ux_sq+uy_sq+uz_sq));
         (D7.f[1])[ke   ] = ConcD*(c1o6*(ae+c1o1)+c1o2*(ux_sq)+vx1*c1o2);
         (D7.f[2])[kw   ] = ConcD*(c1o6*(ae+c1o1)+c1o2*(ux_sq)-vx1*c1o2);
         (D7.f[3])[kn   ] = ConcD*(c1o6*(ae+c1o1)+c1o2*(uy_sq)+vx2*c1o2);
         (D7.f[4])[ks   ] = ConcD*(c1o6*(ae+c1o1)+c1o2*(uy_sq)-vx2*c1o2);
         (D7.f[5])[kt   ] = ConcD*(c1o6*(ae+c1o1)+c1o2*(uz_sq)+vx3*c1o2);
         (D7.f[6])[kb   ] = ConcD*(c1o6*(ae+c1o1)+c1o2*(uz_sq)-vx3*c1o2);
      }
   }
}