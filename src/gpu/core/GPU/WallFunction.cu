#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include <basics/constants/NumericConstants.h>

using namespace vf::basics::constant;
using namespace vf::lbm::dir;


//////////////////////////////////////////////////////////////////////////////
__global__ void WallFunction27(
										  real* vx,
										  real* vy,
										  real* vz,
										  real* DD, 
										  int* k_Q, 
										  real* QQ,
										  unsigned int numberOfBCnodes, 
										  real om1, 
										  unsigned int* neighborX,
										  unsigned int* neighborY,
										  unsigned int* neighborZ,
										  unsigned long long numberOfLBnodes, 
										  bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dP00] = &DD[dP00 * numberOfLBnodes];
      D.f[dM00] = &DD[dM00 * numberOfLBnodes];
      D.f[d0P0] = &DD[d0P0 * numberOfLBnodes];
      D.f[d0M0] = &DD[d0M0 * numberOfLBnodes];
      D.f[d00P] = &DD[d00P * numberOfLBnodes];
      D.f[d00M] = &DD[d00M * numberOfLBnodes];
      D.f[dPP0] = &DD[dPP0 * numberOfLBnodes];
      D.f[dMM0] = &DD[dMM0 * numberOfLBnodes];
      D.f[dPM0] = &DD[dPM0 * numberOfLBnodes];
      D.f[dMP0] = &DD[dMP0 * numberOfLBnodes];
      D.f[dP0P] = &DD[dP0P * numberOfLBnodes];
      D.f[dM0M] = &DD[dM0M * numberOfLBnodes];
      D.f[dP0M] = &DD[dP0M * numberOfLBnodes];
      D.f[dM0P] = &DD[dM0P * numberOfLBnodes];
      D.f[d0PP] = &DD[d0PP * numberOfLBnodes];
      D.f[d0MM] = &DD[d0MM * numberOfLBnodes];
      D.f[d0PM] = &DD[d0PM * numberOfLBnodes];
      D.f[d0MP] = &DD[d0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[dPPP] = &DD[dPPP * numberOfLBnodes];
      D.f[dMMP] = &DD[dMMP * numberOfLBnodes];
      D.f[dPMP] = &DD[dPMP * numberOfLBnodes];
      D.f[dMPP] = &DD[dMPP * numberOfLBnodes];
      D.f[dPPM] = &DD[dPPM * numberOfLBnodes];
      D.f[dMMM] = &DD[dMMM * numberOfLBnodes];
      D.f[dPMM] = &DD[dPMM * numberOfLBnodes];
      D.f[dMPM] = &DD[dMPM * numberOfLBnodes];
   } 
   else
   {
      D.f[dM00] = &DD[dP00 * numberOfLBnodes];
      D.f[dP00] = &DD[dM00 * numberOfLBnodes];
      D.f[d0M0] = &DD[d0P0 * numberOfLBnodes];
      D.f[d0P0] = &DD[d0M0 * numberOfLBnodes];
      D.f[d00M] = &DD[d00P * numberOfLBnodes];
      D.f[d00P] = &DD[d00M * numberOfLBnodes];
      D.f[dMM0] = &DD[dPP0 * numberOfLBnodes];
      D.f[dPP0] = &DD[dMM0 * numberOfLBnodes];
      D.f[dMP0] = &DD[dPM0 * numberOfLBnodes];
      D.f[dPM0] = &DD[dMP0 * numberOfLBnodes];
      D.f[dM0M] = &DD[dP0P * numberOfLBnodes];
      D.f[dP0P] = &DD[dM0M * numberOfLBnodes];
      D.f[dM0P] = &DD[dP0M * numberOfLBnodes];
      D.f[dP0M] = &DD[dM0P * numberOfLBnodes];
      D.f[d0MM] = &DD[d0PP * numberOfLBnodes];
      D.f[d0PP] = &DD[d0MM * numberOfLBnodes];
      D.f[d0MP] = &DD[d0PM * numberOfLBnodes];
      D.f[d0PM] = &DD[d0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[dPPP] = &DD[dMMM * numberOfLBnodes];
      D.f[dMMP] = &DD[dPPM * numberOfLBnodes];
      D.f[dPMP] = &DD[dMPM * numberOfLBnodes];
      D.f[dMPP] = &DD[dPMM * numberOfLBnodes];
      D.f[dPPM] = &DD[dMMP * numberOfLBnodes];
      D.f[dMMM] = &DD[dPPP * numberOfLBnodes];
      D.f[dPMM] = &DD[dMPP * numberOfLBnodes];
      D.f[dMPM] = &DD[dPMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real VeloX = vx[k];
      //real VeloY = vy[k];
      //real VeloZ = vz[k]; //(16.0*(u0*2.0)*bbx*bby*(grid_nx-bbx)*(grid_ny-bby))/(grid_nx*grid_nx*grid_ny*grid_ny)
      ////////////////////////////////////////////////////////////////////////////////
      //real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
      //      *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
      //      *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
      //      *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
      //      *q_dirBSE, *q_dirBNW; 
      //q_dirE   = &QQ[dP00 * numberOfBCnodes];
      //q_dirW   = &QQ[dM00 * numberOfBCnodes];
      //q_dirN   = &QQ[d0P0 * numberOfBCnodes];
      //q_dirS   = &QQ[d0M0 * numberOfBCnodes];
      //q_dirT   = &QQ[d00P * numberOfBCnodes];
      //q_dirB   = &QQ[d00M * numberOfBCnodes];
      //q_dirNE  = &QQ[dPP0 * numberOfBCnodes];
      //q_dirSW  = &QQ[dMM0 * numberOfBCnodes];
      //q_dirSE  = &QQ[dPM0 * numberOfBCnodes];
      //q_dirNW  = &QQ[dMP0 * numberOfBCnodes];
      //q_dirTE  = &QQ[dP0P * numberOfBCnodes];
      //q_dirBW  = &QQ[dM0M * numberOfBCnodes];
      //q_dirBE  = &QQ[dP0M * numberOfBCnodes];
      //q_dirTW  = &QQ[dM0P * numberOfBCnodes];
      //q_dirTN  = &QQ[d0PP * numberOfBCnodes];
      //q_dirBS  = &QQ[d0MM * numberOfBCnodes];
      //q_dirBN  = &QQ[d0PM * numberOfBCnodes];
      //q_dirTS  = &QQ[d0MP * numberOfBCnodes];
      //q_dirTNE = &QQ[dPPP * numberOfBCnodes];
      //q_dirTSW = &QQ[dMMP * numberOfBCnodes];
      //q_dirTSE = &QQ[dPMP * numberOfBCnodes];
      //q_dirTNW = &QQ[dMPP * numberOfBCnodes];
      //q_dirBNE = &QQ[dPPM * numberOfBCnodes];
      //q_dirBSW = &QQ[dMMM * numberOfBCnodes];
      //q_dirBSE = &QQ[dPMM * numberOfBCnodes];
      //q_dirBNW = &QQ[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dP00])[ke   ];
      f_E    = (D.f[dM00])[kw   ];
      f_S    = (D.f[d0P0])[kn   ];
      f_N    = (D.f[d0M0])[ks   ];
      f_B    = (D.f[d00P])[kt   ];
      f_T    = (D.f[d00M])[kb   ];
      f_SW   = (D.f[dPP0])[kne  ];
      f_NE   = (D.f[dMM0])[ksw  ];
      f_NW   = (D.f[dPM0])[kse  ];
      f_SE   = (D.f[dMP0])[knw  ];
      f_BW   = (D.f[dP0P])[kte  ];
      f_TE   = (D.f[dM0M])[kbw  ];
      f_TW   = (D.f[dP0M])[kbe  ];
      f_BE   = (D.f[dM0P])[ktw  ];
      f_BS   = (D.f[d0PP])[ktn  ];
      f_TN   = (D.f[d0MM])[kbs  ];
      f_TS   = (D.f[d0PM])[kbn  ];
      f_BN   = (D.f[d0MP])[kts  ];
      f_BSW  = (D.f[dPPP])[ktne ];
      f_BNE  = (D.f[dMMP])[ktsw ];
      f_BNW  = (D.f[dPMP])[ktse ];
      f_BSE  = (D.f[dMPP])[ktnw ];
      f_TSW  = (D.f[dPPM])[kbne ];
      f_TNE  = (D.f[dMMM])[kbsw ];
      f_TNW  = (D.f[dPMM])[kbse ];
      f_TSE  = (D.f[dMPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      // real vx2, vx3, feq, q;
      real vx1, drho;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]); 

       vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                 (f_E - f_W)) / (c1o1 + drho); 
         

    //   vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
    //              ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
    //              (f_N - f_S)) / (c1o1 + drho); 

    //   vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
    //              (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
    //              (f_T - f_B)) / (c1o1 + drho); 

      //real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (one + drho);

	  real nu = c1o3 * (c1o1 / om1 - c1o2);
	  real qw = c1o1;
	  real uTau = sqrt(nu * (vx1 - VeloX) / qw);

	  if (abs(uTau)/nu>11){
	  uTau = vx1 * 0.41 / (log10(9.8 * uTau * qw / nu));
	  

	  
	  vx[k] = vx1 - uTau * uTau * qw / nu;
	  vx[k] = (vx[k]> 0.05) ? 0.05 : ((vx[k]< -0.05) ? -0.05 : vx[k] );  
	  }
	  else{ vx[k]=c0o1; }
	  //vy[k] = 0.01;							//Test...muss wieder raus
	  //vz[k] = 0.01;							//Test...muss wieder raus

   //   //////////////////////////////////////////////////////////////////////////
   //   if (isEvenTimestep==false)
   //   {
   //      D.f[dP00] = &DD[dP00 * size_Mat];
   //      D.f[dM00] = &DD[dM00 * size_Mat];
   //      D.f[d0P0] = &DD[d0P0 * size_Mat];
   //      D.f[d0M0] = &DD[d0M0 * size_Mat];
   //      D.f[d00P] = &DD[d00P * size_Mat];
   //      D.f[d00M] = &DD[d00M * size_Mat];
   //      D.f[dPP0] = &DD[dPP0 * size_Mat];
   //      D.f[dMM0] = &DD[dMM0 * size_Mat];
   //      D.f[dPM0] = &DD[dPM0 * size_Mat];
   //      D.f[dMP0] = &DD[dMP0 * size_Mat];
   //      D.f[dP0P] = &DD[dP0P * size_Mat];
   //      D.f[dM0M] = &DD[dM0M * size_Mat];
   //      D.f[dP0M] = &DD[dP0M * size_Mat];
   //      D.f[dM0P] = &DD[dM0P * size_Mat];
   //      D.f[d0PP] = &DD[d0PP * size_Mat];
   //      D.f[d0MM] = &DD[d0MM * size_Mat];
   //      D.f[d0PM] = &DD[d0PM * size_Mat];
   //      D.f[d0MP] = &DD[d0MP * size_Mat];
   //      D.f[d000] = &DD[d000 * size_Mat];
   //      D.f[dPPP] = &DD[dPPP * size_Mat];
   //      D.f[dMMP] = &DD[dMMP * size_Mat];
   //      D.f[dPMP] = &DD[dPMP * size_Mat];
   //      D.f[dMPP] = &DD[dMPP * size_Mat];
   //      D.f[dPPM] = &DD[dPPM * size_Mat];
   //      D.f[dMMM] = &DD[dMMM * size_Mat];
   //      D.f[dPMM] = &DD[dPMM * size_Mat];
   //      D.f[dMPM] = &DD[dMPM * size_Mat];
   //   } 
   //   else
   //   {
   //      D.f[dM00] = &DD[dP00 * size_Mat];
   //      D.f[dP00] = &DD[dM00 * size_Mat];
   //      D.f[d0M0] = &DD[d0P0 * size_Mat];
   //      D.f[d0P0] = &DD[d0M0 * size_Mat];
   //      D.f[d00M] = &DD[d00P * size_Mat];
   //      D.f[d00P] = &DD[d00M * size_Mat];
   //      D.f[dMM0] = &DD[dPP0 * size_Mat];
   //      D.f[dPP0] = &DD[dMM0 * size_Mat];
   //      D.f[dMP0] = &DD[dPM0 * size_Mat];
   //      D.f[dPM0] = &DD[dMP0 * size_Mat];
   //      D.f[dM0M] = &DD[dP0P * size_Mat];
   //      D.f[dP0P] = &DD[dM0M * size_Mat];
   //      D.f[dM0P] = &DD[dP0M * size_Mat];
   //      D.f[dP0M] = &DD[dM0P * size_Mat];
   //      D.f[d0MM] = &DD[d0PP * size_Mat];
   //      D.f[d0PP] = &DD[d0MM * size_Mat];
   //      D.f[d0MP] = &DD[d0PM * size_Mat];
   //      D.f[d0PM] = &DD[d0MP * size_Mat];
   //      D.f[d000] = &DD[d000 * size_Mat];
   //      D.f[dPPP] = &DD[dMMM * size_Mat];
   //      D.f[dMMP] = &DD[dPPM * size_Mat];
   //      D.f[dPMP] = &DD[dMPM * size_Mat];
   //      D.f[dMPP] = &DD[dPMM * size_Mat];
   //      D.f[dPPM] = &DD[dMMP * size_Mat];
   //      D.f[dMMM] = &DD[dPPP * size_Mat];
   //      D.f[dPMM] = &DD[dMPP * size_Mat];
   //      D.f[dMPM] = &DD[dPMP * size_Mat];
   //   }
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
   //   //Test
   //   //(D.f[d000])[k]=c1o10;
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	  ////ToDo anders Klammern

   //   q = q_dirE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*( vx1        )*/+c9over2*( vx1        )*( vx1        ) * (one + drho)-cu_sq); 
   //      (D.f[dM00])[kw]=(one-q)/(one+q)*(f_E-f_W+(f_E+f_W-two*feq*om1)/(one-om1))*c1o2+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[dM00])[kw]=zero;
   //   }

   //   q = q_dirW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(-vx1        )*/+c9over2*(-vx1        )*(-vx1        ) * (one + drho)-cu_sq); 
   //      (D.f[dP00])[ke]=(one-q)/(one+q)*(f_W-f_E+(f_W+f_E-two*feq*om1)/(one-om1))*c1o2+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[dP00])[ke]=zero;
   //   }

   //   q = q_dirN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(    vx2     )*/+c9over2*(     vx2    )*(     vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[d0M0])[ks]=(one-q)/(one+q)*(f_N-f_S+(f_N+f_S-two*feq*om1)/(one-om1))*c1o2+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[d0M0])[ks]=zero;
   //   }

   //   q = q_dirS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(   -vx2     )*/+c9over2*(    -vx2    )*(    -vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[d0P0])[kn]=(one-q)/(one+q)*(f_S-f_N+(f_S+f_N-two*feq*om1)/(one-om1))*c1o2+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[d0P0])[kn]=zero;
   //   }

   //   q = q_dirT[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(         vx3)*/+c9over2*(         vx3)*(         vx3) * (one + drho)-cu_sq); 
   //      (D.f[d00M])[kb]=(one-q)/(one+q)*(f_T-f_B+(f_T+f_B-two*feq*om1)/(one-om1))*c1o2+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[d00M])[kb]=one;
   //   }

   //   q = q_dirB[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c2over27* (drho/*+three*(        -vx3)*/+c9over2*(        -vx3)*(        -vx3) * (one + drho)-cu_sq); 
   //      (D.f[d00P])[kt]=(one-q)/(one+q)*(f_B-f_T+(f_B+f_T-two*feq*om1)/(one-om1))*c1o2+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);// - c2over27 * drho;
   //      //(D.f[d00P])[kt]=zero;
   //   }

   //   q = q_dirNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*( vx1+vx2    )*/+c9over2*( vx1+vx2    )*( vx1+vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[dMM0])[ksw]=(one-q)/(one+q)*(f_NE-f_SW+(f_NE+f_SW-two*feq*om1)/(one-om1))*c1o2+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);// - c1over54 * drho;
   //      //(D.f[dMM0])[ksw]=zero;
   //   }

   //   q = q_dirSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(-vx1-vx2    )*/+c9over2*(-vx1-vx2    )*(-vx1-vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[dPP0])[kne]=(one-q)/(one+q)*(f_SW-f_NE+(f_SW+f_NE-two*feq*om1)/(one-om1))*c1o2+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);// - c1over54 * drho;
   //      //(D.f[dPP0])[kne]=zero;
   //   }

   //   q = q_dirSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*( vx1-vx2    )*/+c9over2*( vx1-vx2    )*( vx1-vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[dMP0])[knw]=(one-q)/(one+q)*(f_SE-f_NW+(f_SE+f_NW-two*feq*om1)/(one-om1))*c1o2+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);// - c1over54 * drho;
   //      //(D.f[dMP0])[knw]=zero;
   //   }

   //   q = q_dirNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(-vx1+vx2    )*/+c9over2*(-vx1+vx2    )*(-vx1+vx2    ) * (one + drho)-cu_sq); 
   //      (D.f[dPM0])[kse]=(one-q)/(one+q)*(f_NW-f_SE+(f_NW+f_SE-two*feq*om1)/(one-om1))*c1o2+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);// - c1over54 * drho;
   //      //(D.f[dPM0])[kse]=zero;
   //   }

   //   q = q_dirTE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*( vx1    +vx3)*/+c9over2*( vx1    +vx3)*( vx1    +vx3) * (one + drho)-cu_sq); 
   //      (D.f[dM0M])[kbw]=(one-q)/(one+q)*(f_TE-f_BW+(f_TE+f_BW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[dM0M])[kbw]=zero;
   //   }

   //   q = q_dirBW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(-vx1    -vx3)*/+c9over2*(-vx1    -vx3)*(-vx1    -vx3) * (one + drho)-cu_sq); 
   //      (D.f[dP0P])[kte]=(one-q)/(one+q)*(f_BW-f_TE+(f_BW+f_TE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[dP0P])[kte]=zero;
   //   }

   //   q = q_dirBE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*( vx1    -vx3)*/+c9over2*( vx1    -vx3)*( vx1    -vx3) * (one + drho)-cu_sq); 
   //      (D.f[dM0P])[ktw]=(one-q)/(one+q)*(f_BE-f_TW+(f_BE+f_TW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[dM0P])[ktw]=zero;
   //   }

   //   q = q_dirTW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(-vx1    +vx3)*/+c9over2*(-vx1    +vx3)*(-vx1    +vx3) * (one + drho)-cu_sq); 
   //      (D.f[dP0M])[kbe]=(one-q)/(one+q)*(f_TW-f_BE+(f_TW+f_BE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[dP0M])[kbe]=zero;
   //   }

   //   q = q_dirTN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(     vx2+vx3)*/+c9over2*(     vx2+vx3)*(     vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[d0MM])[kbs]=(one-q)/(one+q)*(f_TN-f_BS+(f_TN+f_BS-two*feq*om1)/(one-om1))*c1o2+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[d0MM])[kbs]=zero;
   //   }

   //   q = q_dirBS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(    -vx2-vx3)*/+c9over2*(    -vx2-vx3)*(    -vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[d0PP])[ktn]=(one-q)/(one+q)*(f_BS-f_TN+(f_BS+f_TN-two*feq*om1)/(one-om1))*c1o2+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[d0PP])[ktn]=zero;
   //   }

   //   q = q_dirBN[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(     vx2-vx3)*/+c9over2*(     vx2-vx3)*(     vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[d0MP])[kts]=(one-q)/(one+q)*(f_BN-f_TS+(f_BN+f_TS-two*feq*om1)/(one-om1))*c1o2+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[d0MP])[kts]=zero;
   //   }

   //   q = q_dirTS[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over54* (drho/*+three*(    -vx2+vx3)*/+c9over2*(    -vx2+vx3)*(    -vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[d0PM])[kbn]=(one-q)/(one+q)*(f_TS-f_BN+(f_TS+f_BN-two*feq*om1)/(one-om1))*c1o2+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);// - c1over54 * drho;
   //      //(D.f[d0PM])[kbn]=zero;
   //   }

   //   q = q_dirTNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*( vx1+vx2+vx3)*/+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[dMMM])[kbsw]=(one-q)/(one+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[dMMM])[kbsw]=zero;
   //   }

   //   q = q_dirBSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*(-vx1-vx2-vx3)*/+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[dPPP])[ktne]=(one-q)/(one+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[dPPP])[ktne]=zero;
   //   }

   //   q = q_dirBNE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*( vx1+vx2-vx3)*/+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[dMMP])[ktsw]=(one-q)/(one+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[dMMP])[ktsw]=zero;
   //   }

   //   q = q_dirTSW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*(-vx1-vx2+vx3)*/+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[dPPM])[kbne]=(one-q)/(one+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[dPPM])[kbne]=zero;
   //   }

   //   q = q_dirTSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*( vx1-vx2+vx3)*/+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[dMPM])[kbnw]=(one-q)/(one+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[dMPM])[kbnw]=zero;
   //   }

   //   q = q_dirBNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*(-vx1+vx2-vx3)*/+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[dPMP])[ktse]=(one-q)/(one+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[dPMP])[ktse]=zero;
   //   }

   //   q = q_dirBSE[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*( vx1-vx2-vx3)*/+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (one + drho)-cu_sq); 
   //      (D.f[dMPP])[ktnw]=(one-q)/(one+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[dMPP])[ktnw]=zero;
   //   }

   //   q = q_dirTNW[k];
   //   if (q>=zero && q<=one)
   //   {
   //      feq=c1over216*(drho/*+three*(-vx1+vx2+vx3)*/+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (one + drho)-cu_sq); 
   //      (D.f[dPMM])[kbse]=(one-q)/(one+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);// - c1over216 * drho;
   //      //(D.f[dPMM])[kbse]=zero;
   //   }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////









