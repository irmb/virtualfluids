#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file SlipBCs27.cu
//! \ingroup GPU
//! \author Martin Schoenherr, Anna Wellmann
//======================================================================================
#include "LBM/LB.h" 
#include "lbm/constants/D3Q27.h"
#include "basics/constants/NumericConstants.h"
#include "LBM/GPUHelperFunctions/KernelUtilities.h"

using namespace vf::basics::constant;
using namespace vf::lbm::dir;
using namespace vf::gpu;

//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDevice27(
    real* DD, 
    int* k_Q, 
    real* QQ,
    unsigned int numberOfBCnodes,
    real om1, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   Distributions27 D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, isEvenTimestep);

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dP00 * numberOfBCnodes];
      q_dirW   = &QQ[dM00 * numberOfBCnodes];
      q_dirN   = &QQ[d0P0 * numberOfBCnodes];
      q_dirS   = &QQ[d0M0 * numberOfBCnodes];
      q_dirT   = &QQ[d00P * numberOfBCnodes];
      q_dirB   = &QQ[d00M * numberOfBCnodes];
      q_dirNE  = &QQ[dPP0 * numberOfBCnodes];
      q_dirSW  = &QQ[dMM0 * numberOfBCnodes];
      q_dirSE  = &QQ[dPM0 * numberOfBCnodes];
      q_dirNW  = &QQ[dMP0 * numberOfBCnodes];
      q_dirTE  = &QQ[dP0P * numberOfBCnodes];
      q_dirBW  = &QQ[dM0M * numberOfBCnodes];
      q_dirBE  = &QQ[dP0M * numberOfBCnodes];
      q_dirTW  = &QQ[dM0P * numberOfBCnodes];
      q_dirTN  = &QQ[d0PP * numberOfBCnodes];
      q_dirBS  = &QQ[d0MM * numberOfBCnodes];
      q_dirBN  = &QQ[d0PM * numberOfBCnodes];
      q_dirTS  = &QQ[d0MP * numberOfBCnodes];
      q_dirTNE = &QQ[dPPP * numberOfBCnodes];
      q_dirTSW = &QQ[dMMP * numberOfBCnodes];
      q_dirTSE = &QQ[dPMP * numberOfBCnodes];
      q_dirTNW = &QQ[dMPP * numberOfBCnodes];
      q_dirBNE = &QQ[dPPM * numberOfBCnodes];
      q_dirBSW = &QQ[dMMM * numberOfBCnodes];
      q_dirBSE = &QQ[dPMM * numberOfBCnodes];
      q_dirBNW = &QQ[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[dP00])[ke   ];
      real f_E    = (D.f[dM00])[kw   ];
      real f_S    = (D.f[d0P0])[kn   ];
      real f_N    = (D.f[d0M0])[ks   ];
      real f_B    = (D.f[d00P])[kt   ];
      real f_T    = (D.f[d00M])[kb   ];
      real f_SW   = (D.f[dPP0])[kne  ];
      real f_NE   = (D.f[dMM0])[ksw  ];
      real f_NW   = (D.f[dPM0])[kse  ];
      real f_SE   = (D.f[dMP0])[knw  ];
      real f_BW   = (D.f[dP0P])[kte  ];
      real f_TE   = (D.f[dM0M])[kbw  ];
      real f_TW   = (D.f[dP0M])[kbe  ];
      real f_BE   = (D.f[dM0P])[ktw  ];
      real f_BS   = (D.f[d0PP])[ktn  ];
      real f_TN   = (D.f[d0MM])[kbs  ];
      real f_TS   = (D.f[d0PM])[kbn  ];
      real f_BN   = (D.f[d0MP])[kts  ];
      real f_BSW  = (D.f[dPPP])[ktne ];
      real f_BNE  = (D.f[dMMP])[ktsw ];
      real f_BNW  = (D.f[dPMP])[ktse ];
      real f_BSE  = (D.f[dMPP])[ktnw ];
      real f_TSW  = (D.f[dPPM])[kbne ];
      real f_TNE  = (D.f[dMMM])[kbsw ];
      real f_TNW  = (D.f[dPMM])[kbse ];
      real f_TSE  = (D.f[dMPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]); 

      vx1    =  ((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W); 
         

      vx2    =   (-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S); 

      vx3    =   ((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      //////////////////////////////////////////////////////////////////////////

      D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, !isEvenTimestep);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[d000])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real fac = c1o1;//c99o100;
      real VeloX = fac*vx1;
      real VeloY = fac*vx2;
      real VeloZ = fac*vx3;
      bool x = false;
      bool y = false;
      bool z = false;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         x = true;
         feq=c2o27* (drho+c3o1*( vx1        )+c9o2*( vx1        )*( vx1        )-cu_sq); 
         (D.f[dM00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-feq*om1)/(c1o1-om1)+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q);
         //(D.f[dM00])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         x = true;
         feq=c2o27* (drho+c3o1*(-vx1        )+c9o2*(-vx1        )*(-vx1        )-cu_sq); 
         (D.f[dP00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-feq*om1)/(c1o1-om1)+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q);
         //(D.f[dP00])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = c0o1;
         VeloZ = fac*vx3;
         y = true;
         feq=c2o27* (drho+c3o1*(    vx2     )+c9o2*(     vx2    )*(     vx2    )-cu_sq); 
         (D.f[d0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-feq*om1)/(c1o1-om1)+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q);
         //(D.f[d0M0])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = c0o1;
         VeloZ = fac*vx3;
         y = true;
         feq=c2o27* (drho+c3o1*(   -vx2     )+c9o2*(    -vx2    )*(    -vx2    )-cu_sq); 
         (D.f[d0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-feq*om1)/(c1o1-om1)+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q);
         //(D.f[d0P0])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = c0o1;
         z = true;
         feq=c2o27* (drho+c3o1*(         vx3)+c9o2*(         vx3)*(         vx3)-cu_sq); 
         (D.f[d00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-feq*om1)/(c1o1-om1)+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q);
         //(D.f[d00M])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = c0o1;
         z = true;
         feq=c2o27* (drho+c3o1*(        -vx3)+c9o2*(        -vx3)*(        -vx3)-cu_sq); 
         (D.f[d00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-feq*om1)/(c1o1-om1)+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q);
         //(D.f[d00P])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*( vx1+vx2    )+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         (D.f[dMM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-feq*om1)/(c1o1-om1)+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
         //(D.f[dMM0])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1-vx2    )+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         (D.f[dPP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-feq*om1)/(c1o1-om1)+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
         //(D.f[dPP0])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*( vx1-vx2    )+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         (D.f[dMP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-feq*om1)/(c1o1-om1)+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
         //(D.f[dMP0])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1+vx2    )+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         (D.f[dPM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-feq*om1)/(c1o1-om1)+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
         //(D.f[dPM0])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*( vx1    +vx3)+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         (D.f[dM0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-feq*om1)/(c1o1-om1)+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
         //(D.f[dM0M])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1    -vx3)+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         (D.f[dP0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-feq*om1)/(c1o1-om1)+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
         //(D.f[dP0P])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*( vx1    -vx3)+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         (D.f[dM0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-feq*om1)/(c1o1-om1)+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
         //(D.f[dM0P])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(-vx1    +vx3)+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         (D.f[dP0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-feq*om1)/(c1o1-om1)+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
         //(D.f[dP0M])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(     vx2+vx3)+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         (D.f[d0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-feq*om1)/(c1o1-om1)+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
         //(D.f[d0MM])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(    -vx2-vx3)+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         (D.f[d0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-feq*om1)/(c1o1-om1)+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
         //(D.f[d0PP])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(     vx2-vx3)+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         (D.f[d0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-feq*om1)/(c1o1-om1)+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
         //(D.f[d0MP])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o54* (drho+c3o1*(    -vx2+vx3)+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         (D.f[d0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-feq*om1)/(c1o1-om1)+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
         //(D.f[d0PM])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1+vx2+vx3)+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         (D.f[dMMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-feq*om1)/(c1o1-om1)+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
         //(D.f[dMMM])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1-vx2-vx3)+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         (D.f[dPPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-feq*om1)/(c1o1-om1)+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
         //(D.f[dPPP])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1+vx2-vx3)+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         (D.f[dMMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-feq*om1)/(c1o1-om1)+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
         //(D.f[dMMP])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1-vx2+vx3)+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         (D.f[dPPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-feq*om1)/(c1o1-om1)+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
         //(D.f[dPPM])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1-vx2+vx3)+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         (D.f[dMPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-feq*om1)/(c1o1-om1)+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
         //(D.f[dMPM])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1+vx2-vx3)+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         (D.f[dPMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-feq*om1)/(c1o1-om1)+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
         //(D.f[dPMP])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*( vx1-vx2-vx3)+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         (D.f[dMPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-feq*om1)/(c1o1-om1)+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
         //(D.f[dMPP])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = fac*vx1;
         VeloY = fac*vx2;
         VeloZ = fac*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         feq=c1o216*(drho+c3o1*(-vx1+vx2+vx3)+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         (D.f[dPMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-feq*om1)/(c1o1-om1)+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
         //(D.f[dPMM])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDeviceComp27(
    real* distributions, 
    int* subgridDistanceIndices, 
    real* subgridDistances,
    unsigned int numberOfBCnodes,
    real omega, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!

   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned nodeIndex = getNodeIndex();

   if(nodeIndex < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[nodeIndex];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[dP00])[ke   ];
      real f_E    = (dist.f[dM00])[kw   ];
      real f_S    = (dist.f[d0P0])[kn   ];
      real f_N    = (dist.f[d0M0])[ks   ];
      real f_B    = (dist.f[d00P])[kt   ];
      real f_T    = (dist.f[d00M])[kb   ];
      real f_SW   = (dist.f[dPP0])[kne  ];
      real f_NE   = (dist.f[dMM0])[ksw  ];
      real f_NW   = (dist.f[dPM0])[kse  ];
      real f_SE   = (dist.f[dMP0])[knw  ];
      real f_BW   = (dist.f[dP0P])[kte  ];
      real f_TE   = (dist.f[dM0M])[kbw  ];
      real f_TW   = (dist.f[dP0M])[kbe  ];
      real f_BE   = (dist.f[dM0P])[ktw  ];
      real f_BS   = (dist.f[d0PP])[ktn  ];
      real f_TN   = (dist.f[d0MM])[kbs  ];
      real f_TS   = (dist.f[d0PM])[kbn  ];
      real f_BN   = (dist.f[d0MP])[kts  ];
      real f_BSW  = (dist.f[dPPP])[ktne ];
      real f_BNE  = (dist.f[dMMP])[ktsw ];
      real f_BNW  = (dist.f[dPMP])[ktse ];
      real f_BSE  = (dist.f[dMPP])[ktnw ];
      real f_TSW  = (dist.f[dPPM])[kbne ];
      real f_TNE  = (dist.f[dMMM])[kbsw ];
      real f_TNW  = (dist.f[dPMM])[kbse ];
      real f_TSE  = (dist.f[dMPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[d000])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real feq, q, velocityLB, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[dP00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloX;
         (dist.f[dM00])[kw] = getInterpolatedDistributionForVeloBC(q, f_E, f_W, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[dM00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloX;
         (dist.f[dP00])[ke] = getInterpolatedDistributionForVeloBC(q, f_W, f_E, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[d0P0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloY;
         (dist.f[d0M0])[ks] = getInterpolatedDistributionForVeloBC(q, f_N, f_S, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[d0M0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloY;
         (dist.f[d0P0])[kn] = getInterpolatedDistributionForVeloBC(q, f_S, f_N, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[d00P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloZ;
         (dist.f[d00M])[kb] = getInterpolatedDistributionForVeloBC(q, f_T, f_B, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[d00M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloZ;
         (dist.f[d00P])[kt] = getInterpolatedDistributionForVeloBC(q, f_B, f_T, feq, omega, velocityBC, c2o27);
      }

      q = (subgridD.q[dPP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloY;
         (dist.f[dMM0])[ksw] = getInterpolatedDistributionForVeloBC(q, f_NE, f_SW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[dMM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloY;
         (dist.f[dPP0])[kne] = getInterpolatedDistributionForVeloBC(q, f_SW, f_NE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[dPM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloY;
         (dist.f[dMP0])[knw] = getInterpolatedDistributionForVeloBC(q, f_SE, f_NW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[dMP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloY;
         (dist.f[dPM0])[kse] = getInterpolatedDistributionForVeloBC(q, f_NW, f_SE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[dP0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloZ;
         (dist.f[dM0M])[kbw] = getInterpolatedDistributionForVeloBC(q, f_TE, f_BW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[dM0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloZ;
         (dist.f[dP0P])[kte] = getInterpolatedDistributionForVeloBC(q, f_BW, f_TE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[dP0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloZ;
         (dist.f[dM0P])[ktw] = getInterpolatedDistributionForVeloBC(q, f_BE, f_TW, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[dM0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloZ;
         (dist.f[dP0M])[kbe] = getInterpolatedDistributionForVeloBC(q, f_TW, f_BE, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[d0PP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY + VeloZ;
         (dist.f[d0MM])[kbs] = getInterpolatedDistributionForVeloBC(q, f_TN, f_BS, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[d0MM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY - VeloZ;
         (dist.f[d0PP])[ktn] = getInterpolatedDistributionForVeloBC(q, f_BS, f_TN, feq, omega, velocityBC, c1o54);
      }


      q = (subgridD.q[d0PM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY - VeloZ;
         (dist.f[d0MP])[kts] = getInterpolatedDistributionForVeloBC(q, f_BN, f_TS, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[d0MP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY + VeloZ;
         (dist.f[d0PM])[kbn] = getInterpolatedDistributionForVeloBC(q, f_TS, f_BN, feq, omega, velocityBC, c1o54);
      }

      q = (subgridD.q[dPPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[dMMM])[kbsw] = getInterpolatedDistributionForVeloBC(q, f_TNE, f_BSW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[dMMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[dPPP])[ktne] = getInterpolatedDistributionForVeloBC(q, f_BSW, f_TNE, feq, omega, velocityBC, c1o216);
      }


      q = (subgridD.q[dPPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[dMMP])[ktsw] = getInterpolatedDistributionForVeloBC(q, f_BNE, f_TSW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[dMMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[dPPM])[kbne] = getInterpolatedDistributionForVeloBC(q, f_TSW, f_BNE, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[dPMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[dMPM])[kbnw] = getInterpolatedDistributionForVeloBC(q, f_TSE, f_BNW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[dMPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[dPMP])[ktse] = getInterpolatedDistributionForVeloBC(q, f_BNW, f_TSE, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[dPMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[dMPP])[ktnw] = getInterpolatedDistributionForVeloBC(q, f_BSE, f_TNW, feq, omega, velocityBC, c1o216);
      }

      q = (subgridD.q[dMPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[dPMM])[kbse] = getInterpolatedDistributionForVeloBC(q, f_TNW, f_BSE, feq, omega, velocityBC, c1o216);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



























//////////////////////////////////////////////////////////////////////////////
__global__ void BBSlipDeviceComp27(
    real* distributions, 
    int* subgridDistanceIndices, 
    real* subgridDistances,
    unsigned int numberOfBCnodes,
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!

   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned nodeIndex = getNodeIndex();

   if(nodeIndex < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist = vf::gpu::getDistributionReferences27(distributions, numberOfLBnodes, isEvenTimestep);
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[nodeIndex];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[dP00])[ke   ];
      real f_E    = (dist.f[dM00])[kw   ];
      real f_S    = (dist.f[d0P0])[kn   ];
      real f_N    = (dist.f[d0M0])[ks   ];
      real f_B    = (dist.f[d00P])[kt   ];
      real f_T    = (dist.f[d00M])[kb   ];
      real f_SW   = (dist.f[dPP0])[kne  ];
      real f_NE   = (dist.f[dMM0])[ksw  ];
      real f_NW   = (dist.f[dPM0])[kse  ];
      real f_SE   = (dist.f[dMP0])[knw  ];
      real f_BW   = (dist.f[dP0P])[kte  ];
      real f_TE   = (dist.f[dM0M])[kbw  ];
      real f_TW   = (dist.f[dP0M])[kbe  ];
      real f_BE   = (dist.f[dM0P])[ktw  ];
      real f_BS   = (dist.f[d0PP])[ktn  ];
      real f_TN   = (dist.f[d0MM])[kbs  ];
      real f_TS   = (dist.f[d0PM])[kbn  ];
      real f_BN   = (dist.f[d0MP])[kts  ];
      real f_BSW  = (dist.f[dPPP])[ktne ];
      real f_BNE  = (dist.f[dMMP])[ktsw ];
      real f_BNW  = (dist.f[dPMP])[ktse ];
      real f_BSE  = (dist.f[dMPP])[ktnw ];
      real f_TSW  = (dist.f[dPPM])[kbne ];
      real f_TNE  = (dist.f[dMMM])[kbsw ];
      real f_TNW  = (dist.f[dPMM])[kbse ];
      real f_TSE  = (dist.f[dMPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[d000])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      // real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!

      dist = vf::gpu::getDistributionReferences27(distributions, numberOfLBnodes, !isEvenTimestep);
      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real q, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[dP00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityBC = VeloX;
         (dist.f[dM00])[kw] = getBounceBackDistributionForVeloBC(f_W, velocityBC, c2o27);
      }

      q = (subgridD.q[dM00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityBC = -VeloX;
         (dist.f[dP00])[ke] = getBounceBackDistributionForVeloBC(f_E, velocityBC, c2o27);
      }

      q = (subgridD.q[d0P0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityBC = VeloY;
         (dist.f[d0M0])[ks] = getBounceBackDistributionForVeloBC(f_S, velocityBC, c2o27);
      }

      q = (subgridD.q[d0M0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityBC = -VeloY;
         (dist.f[d0P0])[kn] = getBounceBackDistributionForVeloBC(f_N, velocityBC, c2o27);
      }

      q = (subgridD.q[d00P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityBC = VeloZ;
         (dist.f[d00M])[kb] = getBounceBackDistributionForVeloBC(f_B, velocityBC, c2o27);
      }

      q = (subgridD.q[d00M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityBC = -VeloZ;
         (dist.f[d00P])[kt] = getBounceBackDistributionForVeloBC(f_T, velocityBC, c2o27);
      }

      q = (subgridD.q[dPP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityBC = VeloX + VeloY;
         (dist.f[dMM0])[ksw] = getBounceBackDistributionForVeloBC(f_SW, velocityBC, c1o54);
      }

      q = (subgridD.q[dMM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityBC = -VeloX - VeloY;
         (dist.f[dPP0])[kne] = getBounceBackDistributionForVeloBC(f_NE, velocityBC, c1o54);
      }

      q = (subgridD.q[dPM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityBC = VeloX - VeloY;
         (dist.f[dMP0])[knw] = getBounceBackDistributionForVeloBC(f_NW, velocityBC, c1o54);
      }

      q = (subgridD.q[dMP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityBC = -VeloX + VeloY;
         (dist.f[dPM0])[kse] = getBounceBackDistributionForVeloBC(f_SE, velocityBC, c1o54);
      }

      q = (subgridD.q[dP0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX + VeloZ;
         (dist.f[dM0M])[kbw] = getBounceBackDistributionForVeloBC(f_BW, velocityBC, c1o54);
      }

      q = (subgridD.q[dM0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

        velocityBC = -VeloX - VeloZ;
        (dist.f[dP0P])[kte] = getBounceBackDistributionForVeloBC(f_TE, velocityBC, c1o54);
      }

      q = (subgridD.q[dP0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX - VeloZ;
         (dist.f[dM0P])[ktw] = getBounceBackDistributionForVeloBC(f_TW, velocityBC, c1o54);
      }

      q = (subgridD.q[dM0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX + VeloZ;
         (dist.f[dP0M])[kbe] = getBounceBackDistributionForVeloBC(f_BE, velocityBC, c1o54);
      }

      q = (subgridD.q[d0PP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloY + VeloZ;
         (dist.f[d0MM])[kbs] = getBounceBackDistributionForVeloBC(f_BS, velocityBC, c1o54);
      }

      q = (subgridD.q[d0MM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloY - VeloZ;
         (dist.f[d0PP])[ktn] = getBounceBackDistributionForVeloBC(f_TN, velocityBC, c1o54);
      }


      q = (subgridD.q[d0PM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloY - VeloZ;
         (dist.f[d0MP])[kts] = getBounceBackDistributionForVeloBC(f_TS, velocityBC, c1o54);
      }

      q = (subgridD.q[d0MP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloY + VeloZ;
         (dist.f[d0PM])[kbn] = getBounceBackDistributionForVeloBC(f_BN, velocityBC, c1o54);
      }

      q = (subgridD.q[dPPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[dMMM])[kbsw] = getBounceBackDistributionForVeloBC(f_TNE, velocityBC, c1o216);
      }

      q = (subgridD.q[dMMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[dPPP])[ktne] = getBounceBackDistributionForVeloBC(f_TNE, velocityBC, c1o216);
      }


      q = (subgridD.q[dPPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[dMMP])[ktsw] = getBounceBackDistributionForVeloBC(f_TSW, velocityBC, c1o216);
      }

      q = (subgridD.q[dMMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[dPPM])[kbne] = getBounceBackDistributionForVeloBC(f_BNE, velocityBC, c1o216);
      }

      q = (subgridD.q[dPMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[dMPM])[kbnw] = getBounceBackDistributionForVeloBC(f_BNW, velocityBC, c1o216);
      }

      q = (subgridD.q[dMPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[dPMP])[ktse] = getBounceBackDistributionForVeloBC(f_TSE, velocityBC, c1o216);
      }

      q = (subgridD.q[dPMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[dMPP])[ktnw] = getBounceBackDistributionForVeloBC(f_TNW, velocityBC, c1o216);
      }

      q = (subgridD.q[dMPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[dPMM])[kbse] = getBounceBackDistributionForVeloBC(f_BSE, velocityBC, c1o216);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////




























////////////////////////////////////////////////////////////////////////////
__global__ void QSlipDeviceComp27TurbViscosity(
    real* distributions, 
    int* subgridDistanceIndices, 
    real* subgridDistances,
    unsigned int numberOfBCnodes,
    real omega, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    real* turbViscosity,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!

   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned nodeIndex = getNodeIndex();

   if(nodeIndex < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[nodeIndex];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[dP00])[ke   ];
      real f_E    = (dist.f[dM00])[kw   ];
      real f_S    = (dist.f[d0P0])[kn   ];
      real f_N    = (dist.f[d0M0])[ks   ];
      real f_B    = (dist.f[d00P])[kt   ];
      real f_T    = (dist.f[d00M])[kb   ];
      real f_SW   = (dist.f[dPP0])[kne  ];
      real f_NE   = (dist.f[dMM0])[ksw  ];
      real f_NW   = (dist.f[dPM0])[kse  ];
      real f_SE   = (dist.f[dMP0])[knw  ];
      real f_BW   = (dist.f[dP0P])[kte  ];
      real f_TE   = (dist.f[dM0M])[kbw  ];
      real f_TW   = (dist.f[dP0M])[kbe  ];
      real f_BE   = (dist.f[dM0P])[ktw  ];
      real f_BS   = (dist.f[d0PP])[ktn  ];
      real f_TN   = (dist.f[d0MM])[kbs  ];
      real f_TS   = (dist.f[d0PM])[kbn  ];
      real f_BN   = (dist.f[d0MP])[kts  ];
      real f_BSW  = (dist.f[dPPP])[ktne ];
      real f_BNE  = (dist.f[dMMP])[ktsw ];
      real f_BNW  = (dist.f[dPMP])[ktse ];
      real f_BSE  = (dist.f[dMPP])[ktnw ];
      real f_TSW  = (dist.f[dPPM])[kbne ];
      real f_TNE  = (dist.f[dMMM])[kbsw ];
      real f_TNW  = (dist.f[dPMM])[kbse ];
      real f_TSE  = (dist.f[dMPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[d000])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - compute local relaxation rate
      //!
      real om_turb = omega / (c1o1 + c3o1* omega* max(c0o1, turbViscosity[indexOfBCnode]) );

      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real feq, q, velocityLB, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[dP00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloX;
         (dist.f[dM00])[kw] = getInterpolatedDistributionForVeloBC(q, f_E, f_W, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[dM00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloX;
         (dist.f[dP00])[ke] = getInterpolatedDistributionForVeloBC(q, f_W, f_E, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[d0P0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloY;
         (dist.f[d0M0])[ks] = getInterpolatedDistributionForVeloBC(q, f_N, f_S, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[d0M0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloY;
         (dist.f[d0P0])[kn] = getInterpolatedDistributionForVeloBC(q, f_S, f_N, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[d00P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloZ;
         (dist.f[d00M])[kb] = getInterpolatedDistributionForVeloBC(q, f_T, f_B, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[d00M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloZ;
         (dist.f[d00P])[kt] = getInterpolatedDistributionForVeloBC(q, f_B, f_T, feq, om_turb, velocityBC, c2o27);
      }

      q = (subgridD.q[dPP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloY;
         (dist.f[dMM0])[ksw] = getInterpolatedDistributionForVeloBC(q, f_NE, f_SW, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[dMM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloY;
         (dist.f[dPP0])[kne] = getInterpolatedDistributionForVeloBC(q, f_SW, f_NE, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[dPM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloY;
         (dist.f[dMP0])[knw] = getInterpolatedDistributionForVeloBC(q, f_SE, f_NW, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[dMP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloY;
         (dist.f[dPM0])[kse] = getInterpolatedDistributionForVeloBC(q, f_NW, f_SE, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[dP0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloZ;
         (dist.f[dM0M])[kbw] = getInterpolatedDistributionForVeloBC(q, f_TE, f_BW, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[dM0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

        velocityLB = -vx1 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloX - VeloZ;
        (dist.f[dP0P])[kte] = getInterpolatedDistributionForVeloBC(q, f_BW, f_TE, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[dP0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloZ;
         (dist.f[dM0P])[ktw] = getInterpolatedDistributionForVeloBC(q, f_BE, f_TW, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[dM0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloZ;
         (dist.f[dP0M])[kbe] = getInterpolatedDistributionForVeloBC(q, f_TW, f_BE, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[d0PP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY + VeloZ;
         (dist.f[d0MM])[kbs] = getInterpolatedDistributionForVeloBC(q, f_TN, f_BS, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[d0MM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY - VeloZ;
         (dist.f[d0PP])[ktn] = getInterpolatedDistributionForVeloBC(q, f_BS, f_TN, feq, om_turb, velocityBC, c1o54);
      }


      q = (subgridD.q[d0PM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY - VeloZ;
         (dist.f[d0MP])[kts] = getInterpolatedDistributionForVeloBC(q, f_BN, f_TS, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[d0MP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY + VeloZ;
         (dist.f[d0PM])[kbn] = getInterpolatedDistributionForVeloBC(q, f_TS, f_BN, feq, om_turb, velocityBC, c1o54);
      }

      q = (subgridD.q[dPPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[dMMM])[kbsw] = getInterpolatedDistributionForVeloBC(q, f_TNE, f_BSW, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[dMMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[dPPP])[ktne] = getInterpolatedDistributionForVeloBC(q, f_BSW, f_TNE, feq, om_turb, velocityBC, c1o216);
      }


      q = (subgridD.q[dPPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[dMMP])[ktsw] = getInterpolatedDistributionForVeloBC(q, f_BNE, f_TSW, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[dMMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[dPPM])[kbne] = getInterpolatedDistributionForVeloBC(q, f_TSW, f_BNE, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[dPMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[dMPM])[kbnw] = getInterpolatedDistributionForVeloBC(q, f_TSE, f_BNW, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[dMPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[dPMP])[ktse] = getInterpolatedDistributionForVeloBC(q, f_BNW, f_TSE, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[dPMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[dMPP])[ktnw] = getInterpolatedDistributionForVeloBC(q, f_BSE, f_TNW, feq, om_turb, velocityBC, c1o216);
      }

      q = (subgridD.q[dMPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[dPMM])[kbse] = getInterpolatedDistributionForVeloBC(q, f_TNW, f_BSE, feq, om_turb, velocityBC, c1o216);
      }
   }
}
////////////////////////////////////////////////////////////////////////////






























////////////////////////////////////////////////////////////////////////////
__global__ void QSlipPressureDeviceComp27TurbViscosity(
    real* distributions, 
    int* subgridDistanceIndices, 
    real* subgridDistances,
    unsigned int numberOfBCnodes,
    real omega, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    real* turbViscosity,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   //! The slip boundary condition is executed in the following steps
   //!
   ////////////////////////////////////////////////////////////////////////////////
   //! - Get node index coordinates from threadIdx, blockIdx, blockDim and gridDim.
   //!
   const unsigned nodeIndex = getNodeIndex();

   if(nodeIndex < numberOfBCnodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //! - Read distributions: style of reading and writing the distributions from/to stored arrays dependent on timestep is based on the esoteric twist algorithm \ref
      //! <a href="https://doi.org/10.3390/computation5020019"><b>[ M. Geier et al. (2017), DOI:10.3390/computation5020019 ]</b></a>
      //!
      Distributions27 dist;
      getPointersToDistributions(dist, distributions, numberOfLBnodes, isEvenTimestep);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local subgrid distances (q's)
      //!
      SubgridDistances27 subgridD;
      getPointersToSubgridDistances(subgridD, subgridDistances, numberOfBCnodes);
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set neighbor indices (necessary for indirect addressing)
      //!
      unsigned int indexOfBCnode  = subgridDistanceIndices[nodeIndex];
      unsigned int kzero= indexOfBCnode;
      unsigned int ke   = indexOfBCnode;
      unsigned int kw   = neighborX[indexOfBCnode];
      unsigned int kn   = indexOfBCnode;
      unsigned int ks   = neighborY[indexOfBCnode];
      unsigned int kt   = indexOfBCnode;
      unsigned int kb   = neighborZ[indexOfBCnode];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = indexOfBCnode;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = indexOfBCnode;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = indexOfBCnode;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = indexOfBCnode;
      unsigned int kbsw = neighborZ[ksw];
      
      ////////////////////////////////////////////////////////////////////////////////
      //! - Set local distributions
      //!
      real f_W    = (dist.f[dP00])[ke   ];
      real f_E    = (dist.f[dM00])[kw   ];
      real f_S    = (dist.f[d0P0])[kn   ];
      real f_N    = (dist.f[d0M0])[ks   ];
      real f_B    = (dist.f[d00P])[kt   ];
      real f_T    = (dist.f[d00M])[kb   ];
      real f_SW   = (dist.f[dPP0])[kne  ];
      real f_NE   = (dist.f[dMM0])[ksw  ];
      real f_NW   = (dist.f[dPM0])[kse  ];
      real f_SE   = (dist.f[dMP0])[knw  ];
      real f_BW   = (dist.f[dP0P])[kte  ];
      real f_TE   = (dist.f[dM0M])[kbw  ];
      real f_TW   = (dist.f[dP0M])[kbe  ];
      real f_BE   = (dist.f[dM0P])[ktw  ];
      real f_BS   = (dist.f[d0PP])[ktn  ];
      real f_TN   = (dist.f[d0MM])[kbs  ];
      real f_TS   = (dist.f[d0PM])[kbn  ];
      real f_BN   = (dist.f[d0MP])[kts  ];
      real f_BSW  = (dist.f[dPPP])[ktne ];
      real f_BNE  = (dist.f[dMMP])[ktsw ];
      real f_BNW  = (dist.f[dPMP])[ktse ];
      real f_BSE  = (dist.f[dMPP])[ktnw ];
      real f_TSW  = (dist.f[dPPM])[kbne ];
      real f_TNE  = (dist.f[dMMM])[kbsw ];
      real f_TNW  = (dist.f[dPMM])[kbse ];
      real f_TSE  = (dist.f[dMPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      //! - Calculate macroscopic quantities
      //!
      real drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                  f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                  f_T + f_B + f_N + f_S + f_E + f_W + ((dist.f[d000])[kzero]); 

      real vx1  = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                   (f_E - f_W)) / (c1o1 + drho);

      real vx2  = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                   ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                   (f_N - f_S)) / (c1o1 + drho);

      real vx3  = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                   (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                   (f_T - f_B)) / (c1o1 + drho);

      real cu_sq = c3o2 * (vx1 * vx1 + vx2 * vx2 + vx3 * vx3) * (c1o1 + drho);

      ////////////////////////////////////////////////////////////////////////////////
      //! - change the pointer to write the results in the correct array
      //!
      getPointersToDistributions(dist, distributions, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////
      //! - compute local relaxation rate
      //!
      real om_turb = omega / (c1o1 + c3o1* omega* max(c0o1, turbViscosity[indexOfBCnode]) );

      ////////////////////////////////////////////////////////////////////////////////
      //! - Multiply the local velocities by the slipLength
      //!
      real slipLength = c1o1;
      real VeloX = slipLength*vx1;
      real VeloY = slipLength*vx2;
      real VeloZ = slipLength*vx3;

      ////////////////////////////////////////////////////////////////////////////////
      //! - Update distributions with subgrid distance (q) between zero and one
      //!
      real feq, q, velocityLB, velocityBC;

      bool x = false;
      bool y = false;
      bool z = false;

      q = (subgridD.q[dP00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)  // only update distribution for q between zero and one
      {
         VeloX = c0o1;
         x = true;

         velocityLB = vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloX;
         (dist.f[dM00])[kw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_E, f_W, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[dM00])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = c0o1;
         x = true;

         velocityLB = -vx1;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloX;
         (dist.f[dP00])[ke] = getInterpolatedDistributionForVeloWithPressureBC(q, f_W, f_E, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[d0P0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloY;
         (dist.f[d0M0])[ks] = getInterpolatedDistributionForVeloWithPressureBC(q, f_N, f_S, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[d0M0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = c0o1;
         y = true;

         velocityLB = -vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloY;
         (dist.f[d0P0])[kn] = getInterpolatedDistributionForVeloWithPressureBC(q, f_S, f_N, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[d00P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = VeloZ;
         (dist.f[d00M])[kb] = getInterpolatedDistributionForVeloWithPressureBC(q, f_T, f_B, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[d00M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = c0o1;
         z = true;

         velocityLB = -vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c2o27);
         velocityBC = -VeloZ;
         (dist.f[d00P])[kt] = getInterpolatedDistributionForVeloWithPressureBC(q, f_B, f_T, feq, om_turb, drho, velocityBC, c2o27);
      }

      q = (subgridD.q[dPP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloY;
         (dist.f[dMM0])[ksw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_NE, f_SW, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[dMM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX - VeloY;
         (dist.f[dPP0])[kne] = getInterpolatedDistributionForVeloWithPressureBC(q, f_SW, f_NE, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[dPM0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = vx1 - vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloY;
         (dist.f[dMP0])[knw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_SE, f_NW, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[dMP0])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;

         velocityLB = -vx1 + vx2;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloY;
         (dist.f[dPM0])[kse] = getInterpolatedDistributionForVeloWithPressureBC(q, f_NW, f_SE, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[dP0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX + VeloZ;
         (dist.f[dM0M])[kbw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TE, f_BW, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[dM0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
        VeloX = slipLength*vx1;
        VeloZ = slipLength*vx3;
        if (x == true) VeloX = c0o1;
        if (z == true) VeloZ = c0o1;

        velocityLB = -vx1 - vx3;
        feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
        velocityBC = -VeloX - VeloZ;
        (dist.f[dP0P])[kte] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BW, f_TE, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[dP0M])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx1 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloX - VeloZ;
         (dist.f[dM0P])[ktw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BE, f_TW, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[dM0P])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx1 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloX + VeloZ;
         (dist.f[dP0M])[kbe] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TW, f_BE, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[d0PP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY + VeloZ;
         (dist.f[d0MM])[kbs] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TN, f_BS, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[d0MM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY - VeloZ;
         (dist.f[d0PP])[ktn] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BS, f_TN, feq, om_turb, drho, velocityBC, c1o54);
      }


      q = (subgridD.q[d0PM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = VeloY - VeloZ;
         (dist.f[d0MP])[kts] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BN, f_TS, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[d0MP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;

         velocityLB = -vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o54);
         velocityBC = -VeloY + VeloZ;
         (dist.f[d0PM])[kbn] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TS, f_BN, feq, om_turb, drho, velocityBC, c1o54);
      }

      q = (subgridD.q[dPPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY + VeloZ;
         (dist.f[dMMM])[kbsw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TNE, f_BSW, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[dMMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY - VeloZ;
         (dist.f[dPPP])[ktne] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BSW, f_TNE, feq, om_turb, drho, velocityBC, c1o216);
      }


      q = (subgridD.q[dPPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX + VeloY - VeloZ;
         (dist.f[dMMP])[ktsw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BNE, f_TSW, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[dMMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX - VeloY + VeloZ;
         (dist.f[dPPM])[kbne] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TSW, f_BNE, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[dPMP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY + VeloZ;
         (dist.f[dMPM])[kbnw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TSE, f_BNW, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[dMPM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY - VeloZ;
         (dist.f[dPMP])[ktse] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BNW, f_TSE, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[dPMM])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = vx1 - vx2 - vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = VeloX - VeloY - VeloZ;
         (dist.f[dMPP])[ktnw] = getInterpolatedDistributionForVeloWithPressureBC(q, f_BSE, f_TNW, feq, om_turb, drho, velocityBC, c1o216);
      }

      q = (subgridD.q[dMPP])[nodeIndex];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = slipLength*vx1;
         VeloY = slipLength*vx2;
         VeloZ = slipLength*vx3;
         if (x == true) VeloX = c0o1;
         if (y == true) VeloY = c0o1;
         if (z == true) VeloZ = c0o1;
         velocityLB = -vx1 + vx2 + vx3;
         feq = getEquilibriumForBC(drho, velocityLB, cu_sq, c1o216);
         velocityBC = -VeloX + VeloY + VeloZ;
         (dist.f[dPMM])[kbse] = getInterpolatedDistributionForVeloWithPressureBC(q, f_TNW, f_BSE, feq, om_turb, drho, velocityBC, c1o216);
      }
   }
}

// __global__ void QSlipDeviceComp27TurbViscosity(real* DD, 
//                                              int* k_Q, 
//                                              real* QQ,
//                                              unsigned int numberOfBCnodes,
//                                              real om1, 
//                                              unsigned int* neighborX,
//                                              unsigned int* neighborY,
//                                              unsigned int* neighborZ,
//                                   real* turbViscosity,
//                                              unsigned int size_Mat, 
//                                              bool isEvenTimestep)
// {
//    Distributions27 D;
//    if (isEvenTimestep==true)
//    {
//       D.f[dP00] = &DD[dP00 * size_Mat];
//       D.f[dM00] = &DD[dM00 * size_Mat];
//       D.f[d0P0] = &DD[d0P0 * size_Mat];
//       D.f[d0M0] = &DD[d0M0 * size_Mat];
//       D.f[d00P] = &DD[d00P * size_Mat];
//       D.f[d00M] = &DD[d00M * size_Mat];
//       D.f[dPP0] = &DD[dPP0 * size_Mat];
//       D.f[dMM0] = &DD[dMM0 * size_Mat];
//       D.f[dPM0] = &DD[dPM0 * size_Mat];
//       D.f[dMP0] = &DD[dMP0 * size_Mat];
//       D.f[dP0P] = &DD[dP0P * size_Mat];
//       D.f[dM0M] = &DD[dM0M * size_Mat];
//       D.f[dP0M] = &DD[dP0M * size_Mat];
//       D.f[dM0P] = &DD[dM0P * size_Mat];
//       D.f[d0PP] = &DD[d0PP * size_Mat];
//       D.f[d0MM] = &DD[d0MM * size_Mat];
//       D.f[d0PM] = &DD[d0PM * size_Mat];
//       D.f[d0MP] = &DD[d0MP * size_Mat];
//       D.f[d000] = &DD[d000 * size_Mat];
//       D.f[dPPP] = &DD[dPPP * size_Mat];
//       D.f[dMMP] = &DD[dMMP * size_Mat];
//       D.f[dPMP] = &DD[dPMP * size_Mat];
//       D.f[dMPP] = &DD[dMPP * size_Mat];
//       D.f[dPPM] = &DD[dPPM * size_Mat];
//       D.f[dMMM] = &DD[dMMM * size_Mat];
//       D.f[dPMM] = &DD[dPMM * size_Mat];
//       D.f[dMPM] = &DD[dMPM * size_Mat];
//    } 
//    else
//    {
//       D.f[dM00] = &DD[dP00 * size_Mat];
//       D.f[dP00] = &DD[dM00 * size_Mat];
//       D.f[d0M0] = &DD[d0P0 * size_Mat];
//       D.f[d0P0] = &DD[d0M0 * size_Mat];
//       D.f[d00M] = &DD[d00P * size_Mat];
//       D.f[d00P] = &DD[d00M * size_Mat];
//       D.f[dMM0] = &DD[dPP0 * size_Mat];
//       D.f[dPP0] = &DD[dMM0 * size_Mat];
//       D.f[dMP0] = &DD[dPM0 * size_Mat];
//       D.f[dPM0] = &DD[dMP0 * size_Mat];
//       D.f[dM0M] = &DD[dP0P * size_Mat];
//       D.f[dP0P] = &DD[dM0M * size_Mat];
//       D.f[dM0P] = &DD[dP0M * size_Mat];
//       D.f[dP0M] = &DD[dM0P * size_Mat];
//       D.f[d0MM] = &DD[d0PP * size_Mat];
//       D.f[d0PP] = &DD[d0MM * size_Mat];
//       D.f[d0MP] = &DD[d0PM * size_Mat];
//       D.f[d0PM] = &DD[d0MP * size_Mat];
//       D.f[d000] = &DD[d000 * size_Mat];
//       D.f[dPPP] = &DD[dMMM * size_Mat];
//       D.f[dMMP] = &DD[dPPM * size_Mat];
//       D.f[dPMP] = &DD[dMPM * size_Mat];
//       D.f[dMPP] = &DD[dPMM * size_Mat];
//       D.f[dPPM] = &DD[dMMP * size_Mat];
//       D.f[dMMM] = &DD[dPPP * size_Mat];
//       D.f[dPMM] = &DD[dMPP * size_Mat];
//       D.f[dMPM] = &DD[dPMP * size_Mat];
//    }
//    ////////////////////////////////////////////////////////////////////////////////
//    const unsigned  x = threadIdx.x;  // Globaler x-Index 
//    const unsigned  y = blockIdx.x;   // Globaler y-Index 
//    const unsigned  z = blockIdx.y;   // Globaler z-Index 

//    const unsigned nx = blockDim.x;
//    const unsigned ny = gridDim.x;

//    const unsigned k = nx*(ny*z + y) + x;
//    //////////////////////////////////////////////////////////////////////////

//    if(k<numberOfBCnodes)
//    {
//       ////////////////////////////////////////////////////////////////////////////////
//       real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
//             *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
//             *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
//             *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
//             *q_dirBSE, *q_dirBNW; 
//       q_dirE   = &QQ[dP00 * numberOfBCnodes];
//       q_dirW   = &QQ[dM00 * numberOfBCnodes];
//       q_dirN   = &QQ[d0P0 * numberOfBCnodes];
//       q_dirS   = &QQ[d0M0 * numberOfBCnodes];
//       q_dirT   = &QQ[d00P * numberOfBCnodes];
//       q_dirB   = &QQ[d00M * numberOfBCnodes];
//       q_dirNE  = &QQ[dPP0 * numberOfBCnodes];
//       q_dirSW  = &QQ[dMM0 * numberOfBCnodes];
//       q_dirSE  = &QQ[dPM0 * numberOfBCnodes];
//       q_dirNW  = &QQ[dMP0 * numberOfBCnodes];
//       q_dirTE  = &QQ[dP0P * numberOfBCnodes];
//       q_dirBW  = &QQ[dM0M * numberOfBCnodes];
//       q_dirBE  = &QQ[dP0M * numberOfBCnodes];
//       q_dirTW  = &QQ[dM0P * numberOfBCnodes];
//       q_dirTN  = &QQ[d0PP * numberOfBCnodes];
//       q_dirBS  = &QQ[d0MM * numberOfBCnodes];
//       q_dirBN  = &QQ[d0PM * numberOfBCnodes];
//       q_dirTS  = &QQ[d0MP * numberOfBCnodes];
//       q_dirTNE = &QQ[dPPP * numberOfBCnodes];
//       q_dirTSW = &QQ[dMMP * numberOfBCnodes];
//       q_dirTSE = &QQ[dPMP * numberOfBCnodes];
//       q_dirTNW = &QQ[dMPP * numberOfBCnodes];
//       q_dirBNE = &QQ[dPPM * numberOfBCnodes];
//       q_dirBSW = &QQ[dMMM * numberOfBCnodes];
//       q_dirBSE = &QQ[dPMM * numberOfBCnodes];
//       q_dirBNW = &QQ[dMPM * numberOfBCnodes];
//       ////////////////////////////////////////////////////////////////////////////////
//       //index
//       unsigned int KQK  = k_Q[k];
//       unsigned int kzero= KQK;
//       unsigned int ke   = KQK;
//       unsigned int kw   = neighborX[KQK];
//       unsigned int kn   = KQK;
//       unsigned int ks   = neighborY[KQK];
//       unsigned int kt   = KQK;
//       unsigned int kb   = neighborZ[KQK];
//       unsigned int ksw  = neighborY[kw];
//       unsigned int kne  = KQK;
//       unsigned int kse  = ks;
//       unsigned int knw  = kw;
//       unsigned int kbw  = neighborZ[kw];
//       unsigned int kte  = KQK;
//       unsigned int kbe  = kb;
//       unsigned int ktw  = kw;
//       unsigned int kbs  = neighborZ[ks];
//       unsigned int ktn  = KQK;
//       unsigned int kbn  = kb;
//       unsigned int kts  = ks;
//       unsigned int ktse = ks;
//       unsigned int kbnw = kbw;
//       unsigned int ktnw = kw;
//       unsigned int kbse = kbs;
//       unsigned int ktsw = ksw;
//       unsigned int kbne = kb;
//       unsigned int ktne = KQK;
//       unsigned int kbsw = neighborZ[ksw];
      
//       ////////////////////////////////////////////////////////////////////////////////
//       real f_W    = (D.f[dP00])[ke   ];
//       real f_E    = (D.f[dM00])[kw   ];
//       real f_S    = (D.f[d0P0])[kn   ];
//       real f_N    = (D.f[d0M0])[ks   ];
//       real f_B    = (D.f[d00P])[kt   ];
//       real f_T    = (D.f[d00M])[kb   ];
//       real f_SW   = (D.f[dPP0])[kne  ];
//       real f_NE   = (D.f[dMM0])[ksw  ];
//       real f_NW   = (D.f[dPM0])[kse  ];
//       real f_SE   = (D.f[dMP0])[knw  ];
//       real f_BW   = (D.f[dP0P])[kte  ];
//       real f_TE   = (D.f[dM0M])[kbw  ];
//       real f_TW   = (D.f[dP0M])[kbe  ];
//       real f_BE   = (D.f[dM0P])[ktw  ];
//       real f_BS   = (D.f[d0PP])[ktn  ];
//       real f_TN   = (D.f[d0MM])[kbs  ];
//       real f_TS   = (D.f[d0PM])[kbn  ];
//       real f_BN   = (D.f[d0MP])[kts  ];
//       real f_BSW  = (D.f[dPPP])[ktne ];
//       real f_BNE  = (D.f[dMMP])[ktsw ];
//       real f_BNW  = (D.f[dPMP])[ktse ];
//       real f_BSE  = (D.f[dMPP])[ktnw ];
//       real f_TSW  = (D.f[dPPM])[kbne ];
//       real f_TNE  = (D.f[dMMM])[kbsw ];
//       real f_TNW  = (D.f[dPMM])[kbse ];
//       real f_TSE  = (D.f[dMPM])[kbnw ];
//       ////////////////////////////////////////////////////////////////////////////////
//       real vx1, vx2, vx3, drho, feq, q;
//       drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
//                 f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
//                 f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]); 

//       vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
//                 ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
//                 (f_E - f_W)) / (c1o1 + drho); 
         

//       vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
//                  ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
//                  (f_N - f_S)) / (c1o1 + drho); 

//       vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
//                  (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
//                  (f_T - f_B)) / (c1o1 + drho); 

//       real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

//       //////////////////////////////////////////////////////////////////////////
//       if (isEvenTimestep==false)
//       {
//          D.f[dP00] = &DD[dP00 * size_Mat];
//          D.f[dM00] = &DD[dM00 * size_Mat];
//          D.f[d0P0] = &DD[d0P0 * size_Mat];
//          D.f[d0M0] = &DD[d0M0 * size_Mat];
//          D.f[d00P] = &DD[d00P * size_Mat];
//          D.f[d00M] = &DD[d00M * size_Mat];
//          D.f[dPP0] = &DD[dPP0 * size_Mat];
//          D.f[dMM0] = &DD[dMM0 * size_Mat];
//          D.f[dPM0] = &DD[dPM0 * size_Mat];
//          D.f[dMP0] = &DD[dMP0 * size_Mat];
//          D.f[dP0P] = &DD[dP0P * size_Mat];
//          D.f[dM0M] = &DD[dM0M * size_Mat];
//          D.f[dP0M] = &DD[dP0M * size_Mat];
//          D.f[dM0P] = &DD[dM0P * size_Mat];
//          D.f[d0PP] = &DD[d0PP * size_Mat];
//          D.f[d0MM] = &DD[d0MM * size_Mat];
//          D.f[d0PM] = &DD[d0PM * size_Mat];
//          D.f[d0MP] = &DD[d0MP * size_Mat];
//          D.f[d000] = &DD[d000 * size_Mat];
//          D.f[dPPP] = &DD[dPPP * size_Mat];
//          D.f[dMMP] = &DD[dMMP * size_Mat];
//          D.f[dPMP] = &DD[dPMP * size_Mat];
//          D.f[dMPP] = &DD[dMPP * size_Mat];
//          D.f[dPPM] = &DD[dPPM * size_Mat];
//          D.f[dMMM] = &DD[dMMM * size_Mat];
//          D.f[dPMM] = &DD[dPMM * size_Mat];
//          D.f[dMPM] = &DD[dMPM * size_Mat];
//       } 
//       else
//       {
//          D.f[dM00] = &DD[dP00 * size_Mat];
//          D.f[dP00] = &DD[dM00 * size_Mat];
//          D.f[d0M0] = &DD[d0P0 * size_Mat];
//          D.f[d0P0] = &DD[d0M0 * size_Mat];
//          D.f[d00M] = &DD[d00P * size_Mat];
//          D.f[d00P] = &DD[d00M * size_Mat];
//          D.f[dMM0] = &DD[dPP0 * size_Mat];
//          D.f[dPP0] = &DD[dMM0 * size_Mat];
//          D.f[dMP0] = &DD[dPM0 * size_Mat];
//          D.f[dPM0] = &DD[dMP0 * size_Mat];
//          D.f[dM0M] = &DD[dP0P * size_Mat];
//          D.f[dP0P] = &DD[dM0M * size_Mat];
//          D.f[dM0P] = &DD[dP0M * size_Mat];
//          D.f[dP0M] = &DD[dM0P * size_Mat];
//          D.f[d0MM] = &DD[d0PP * size_Mat];
//          D.f[d0PP] = &DD[d0MM * size_Mat];
//          D.f[d0MP] = &DD[d0PM * size_Mat];
//          D.f[d0PM] = &DD[d0MP * size_Mat];
//          D.f[d000] = &DD[d000 * size_Mat];
//          D.f[dPPP] = &DD[dMMM * size_Mat];
//          D.f[dMMP] = &DD[dPPM * size_Mat];
//          D.f[dPMP] = &DD[dMPM * size_Mat];
//          D.f[dMPP] = &DD[dPMM * size_Mat];
//          D.f[dPPM] = &DD[dMMP * size_Mat];
//          D.f[dMMM] = &DD[dPPP * size_Mat];
//          D.f[dPMM] = &DD[dMPP * size_Mat];
//          D.f[dMPM] = &DD[dPMP * size_Mat];
//       }
//       ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//       //Test
//       //(D.f[d000])[k]=c1o10;
//       ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//       real om_turb = om1 / (c1o1 + c3o1*om1*max(c0o1, turbViscosity[k_Q[k]]));
     
//      real fac = c1o1;//c99o100;
//       real VeloX = fac*vx1;
//       real VeloY = fac*vx2;
//       real VeloZ = fac*vx3;
//       bool x = false;
//       bool y = false;
//       bool z = false;

//       q = q_dirE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = c0o1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          x = true;
//          feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
//          (D.f[dM00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cu_sq); 
//          //(D.f[dM00])[kw]=(one-q)/(one+q)*(f_E-feq*om1)/(one-om1)+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);
//          //(D.f[dM00])[kw]=zero;
//       }

//       q = q_dirW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = c0o1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          x = true;
//          feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
//          (D.f[dP00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cu_sq); 
//          //(D.f[dP00])[ke]=(one-q)/(one+q)*(f_W-feq*om_turb)/(one-om_turb)+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);
//          //(D.f[dP00])[ke]=zero;
//       }

//       q = q_dirN[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = c0o1;
//          VeloZ = fac*vx3;
//          y = true;
//          feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[d0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cu_sq); 
//          //(D.f[d0M0])[ks]=(one-q)/(one+q)*(f_N-feq*om_turb)/(one-om_turb)+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);
//          //(D.f[d0M0])[ks]=zero;
//       }

//       q = q_dirS[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = c0o1;
//          VeloZ = fac*vx3;
//          y = true;
//          feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[d0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cu_sq); 
//          //(D.f[d0P0])[kn]=(one-q)/(one+q)*(f_S-feq*om_turb)/(one-om_turb)+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);
//          //(D.f[d0P0])[kn]=zero;
//       }

//       q = q_dirT[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = c0o1;
//          z = true;
//          feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[d00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cu_sq); 
//          //(D.f[d00M])[kb]=(one-q)/(one+q)*(f_T-feq*om_turb)/(one-om_turb)+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);
//          //(D.f[d00M])[kb]=one;
//       }

//       q = q_dirB[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = c0o1;
//          z = true;
//          feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[d00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
//          //feq=c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cu_sq); 
//          //(D.f[d00P])[kt]=(one-q)/(one+q)*(f_B-feq*om_turb)/(one-om_turb)+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);
//          //(D.f[d00P])[kt]=zero;
//       }

//       q = q_dirNE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[dMM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
//          //(D.f[dMM0])[ksw]=(one-q)/(one+q)*(f_NE-feq*om_turb)/(one-om_turb)+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);
//          //(D.f[dMM0])[ksw]=zero;
//       }

//       q = q_dirSW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[dPP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
//          //(D.f[dPP0])[kne]=(one-q)/(one+q)*(f_SW-feq*om_turb)/(one-om_turb)+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);
//          //(D.f[dPP0])[kne]=zero;
//       }

//       q = q_dirSE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[dMP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
//          //(D.f[dMP0])[knw]=(one-q)/(one+q)*(f_SE-feq*om_turb)/(one-om_turb)+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);
//          //(D.f[dMP0])[knw]=zero;
//       }

//       q = q_dirNW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
//          (D.f[dPM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
//          //(D.f[dPM0])[kse]=(one-q)/(one+q)*(f_NW-feq*om_turb)/(one-om_turb)+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);
//          //(D.f[dPM0])[kse]=zero;
//       }

//       q = q_dirTE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (z == true) VeloZ = c0o1;
//       //  if (k==10000) printf("AFTER x: %u \t  y: %u \t z: %u \n  VeloX: %f \t VeloY: %f \t VeloZ: %f \n\n", x,y,z, VeloX,VeloY,VeloZ);
//          feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dM0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
//          //(D.f[dM0M])[kbw]=(one-q)/(one+q)*(f_TE-feq*om_turb)/(one-om_turb)+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);
//          //(D.f[dM0M])[kbw]=zero;
//       }

//       q = q_dirBW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dP0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
//          //(D.f[dP0P])[kte]=(one-q)/(one+q)*(f_BW-feq*om_turb)/(one-om_turb)+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);
//          //(D.f[dP0P])[kte]=zero;
//       }

//       q = q_dirBE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dM0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
//          //(D.f[dM0P])[ktw]=(one-q)/(one+q)*(f_BE-feq*om_turb)/(one-om_turb)+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);
//          //(D.f[dM0P])[ktw]=zero;
//       }

//       q = q_dirTW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dP0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
//          //(D.f[dP0M])[kbe]=(one-q)/(one+q)*(f_TW-feq*om_turb)/(one-om_turb)+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);
//          //(D.f[dP0M])[kbe]=zero;
//       }

//       q = q_dirTN[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[d0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
//          //(D.f[d0MM])[kbs]=(one-q)/(one+q)*(f_TN-feq*om_turb)/(one-om_turb)+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);
//          //(D.f[d0MM])[kbs]=zero;
//       }

//       q = q_dirBS[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[d0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
//          //(D.f[d0PP])[ktn]=(one-q)/(one+q)*(f_BS-feq*om_turb)/(one-om_turb)+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);
//          //(D.f[d0PP])[ktn]=zero;
//       }

//       q = q_dirBN[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[d0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
//          //(D.f[d0MP])[kts]=(one-q)/(one+q)*(f_BN-feq*om_turb)/(one-om_turb)+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);
//          //(D.f[d0MP])[kts]=zero;
//       }

//       q = q_dirTS[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[d0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
//          //feq=c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
//          //(D.f[d0PM])[kbn]=(one-q)/(one+q)*(f_TS-feq*om_turb)/(one-om_turb)+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);
//          //(D.f[d0PM])[kbn]=zero;
//       }

//       q = q_dirTNE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dMMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
//          //(D.f[dMMM])[kbsw]=(one-q)/(one+q)*(f_TNE-feq*om_turb)/(one-om_turb)+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);
//          //(D.f[dMMM])[kbsw]=zero;
//       }

//       q = q_dirBSW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dPPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
//          //(D.f[dPPP])[ktne]=(one-q)/(one+q)*(f_BSW-feq*om_turb)/(one-om_turb)+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);
//          //(D.f[dPPP])[ktne]=zero;
//       }

//       q = q_dirBNE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dMMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
//          //(D.f[dMMP])[ktsw]=(one-q)/(one+q)*(f_BNE-feq*om_turb)/(one-om_turb)+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);
//          //(D.f[dMMP])[ktsw]=zero;
//       }

//       q = q_dirTSW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dPPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
//          //(D.f[dPPM])[kbne]=(one-q)/(one+q)*(f_TSW-feq*om_turb)/(one-om_turb)+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);
//          //(D.f[dPPM])[kbne]=zero;
//       }

//       q = q_dirTSE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dMPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
//          //(D.f[dMPM])[kbnw]=(one-q)/(one+q)*(f_TSE-feq*om_turb)/(one-om_turb)+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);
//          //(D.f[dMPM])[kbnw]=zero;
//       }

//       q = q_dirBNW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dPMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
//          //(D.f[dPMP])[ktse]=(one-q)/(one+q)*(f_BNW-feq*om_turb)/(one-om_turb)+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);
//          //(D.f[dPMP])[ktse]=zero;
//       }

//       q = q_dirBSE[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dMPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
//          //(D.f[dMPP])[ktnw]=(one-q)/(one+q)*(f_BSE-feq*om_turb)/(one-om_turb)+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);
//          //(D.f[dMPP])[ktnw]=zero;
//       }

//       q = q_dirTNW[k];
//       if (q>=c0o1 && q<=c1o1)
//       {
//          VeloX = fac*vx1;
//          VeloY = fac*vx2;
//          VeloZ = fac*vx3;
//          if (x == true) VeloX = c0o1;
//          if (y == true) VeloY = c0o1;
//          if (z == true) VeloZ = c0o1;
//          feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
//          (D.f[dPMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om_turb)/(c1o1-om_turb))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
//          //feq=c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
//          //(D.f[dPMM])[kbse]=(one-q)/(one+q)*(f_TNW-feq*om_turb)/(one-om_turb)+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);
//          //(D.f[dPMM])[kbse]=zero;
//       }
//    }
// }






































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipGeomDeviceComp27(
    real* DD, 
    int* k_Q, 
    real* QQ,
    unsigned int  numberOfBCnodes,
    real om1, 
    real* NormalX,
    real* NormalY,
    real* NormalZ,
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dP00] = &DD[dP00 * numberOfLBnodes];
      D.f[dM00] = &DD[dM00 * numberOfLBnodes];
      D.f[d0P0] = &DD[d0P0 * numberOfLBnodes];
      D.f[d0M0] = &DD[d0M0 * numberOfLBnodes];
      D.f[d00P] = &DD[d00P * numberOfLBnodes];
      D.f[d00M] = &DD[d00M * numberOfLBnodes];
      D.f[dPP0] = &DD[dPP0 * numberOfLBnodes];
      D.f[dMM0] = &DD[dMM0 * numberOfLBnodes];
      D.f[dPM0] = &DD[dPM0 * numberOfLBnodes];
      D.f[dMP0] = &DD[dMP0 * numberOfLBnodes];
      D.f[dP0P] = &DD[dP0P * numberOfLBnodes];
      D.f[dM0M] = &DD[dM0M * numberOfLBnodes];
      D.f[dP0M] = &DD[dP0M * numberOfLBnodes];
      D.f[dM0P] = &DD[dM0P * numberOfLBnodes];
      D.f[d0PP] = &DD[d0PP * numberOfLBnodes];
      D.f[d0MM] = &DD[d0MM * numberOfLBnodes];
      D.f[d0PM] = &DD[d0PM * numberOfLBnodes];
      D.f[d0MP] = &DD[d0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[dPPP] = &DD[dPPP * numberOfLBnodes];
      D.f[dMMP] = &DD[dMMP * numberOfLBnodes];
      D.f[dPMP] = &DD[dPMP * numberOfLBnodes];
      D.f[dMPP] = &DD[dMPP * numberOfLBnodes];
      D.f[dPPM] = &DD[dPPM * numberOfLBnodes];
      D.f[dMMM] = &DD[dMMM * numberOfLBnodes];
      D.f[dPMM] = &DD[dPMM * numberOfLBnodes];
      D.f[dMPM] = &DD[dMPM * numberOfLBnodes];
   } 
   else
   {
      D.f[dM00] = &DD[dP00 * numberOfLBnodes];
      D.f[dP00] = &DD[dM00 * numberOfLBnodes];
      D.f[d0M0] = &DD[d0P0 * numberOfLBnodes];
      D.f[d0P0] = &DD[d0M0 * numberOfLBnodes];
      D.f[d00M] = &DD[d00P * numberOfLBnodes];
      D.f[d00P] = &DD[d00M * numberOfLBnodes];
      D.f[dMM0] = &DD[dPP0 * numberOfLBnodes];
      D.f[dPP0] = &DD[dMM0 * numberOfLBnodes];
      D.f[dMP0] = &DD[dPM0 * numberOfLBnodes];
      D.f[dPM0] = &DD[dMP0 * numberOfLBnodes];
      D.f[dM0M] = &DD[dP0P * numberOfLBnodes];
      D.f[dP0P] = &DD[dM0M * numberOfLBnodes];
      D.f[dM0P] = &DD[dP0M * numberOfLBnodes];
      D.f[dP0M] = &DD[dM0P * numberOfLBnodes];
      D.f[d0MM] = &DD[d0PP * numberOfLBnodes];
      D.f[d0PP] = &DD[d0MM * numberOfLBnodes];
      D.f[d0MP] = &DD[d0PM * numberOfLBnodes];
      D.f[d0PM] = &DD[d0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[dPPP] = &DD[dMMM * numberOfLBnodes];
      D.f[dMMP] = &DD[dPPM * numberOfLBnodes];
      D.f[dPMP] = &DD[dMPM * numberOfLBnodes];
      D.f[dMPP] = &DD[dPMM * numberOfLBnodes];
      D.f[dPPM] = &DD[dMMP * numberOfLBnodes];
      D.f[dMMM] = &DD[dPPP * numberOfLBnodes];
      D.f[dPMM] = &DD[dMPP * numberOfLBnodes];
      D.f[dMPM] = &DD[dPMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dP00 * numberOfBCnodes];
      q_dirW   = &QQ[dM00 * numberOfBCnodes];
      q_dirN   = &QQ[d0P0 * numberOfBCnodes];
      q_dirS   = &QQ[d0M0 * numberOfBCnodes];
      q_dirT   = &QQ[d00P * numberOfBCnodes];
      q_dirB   = &QQ[d00M * numberOfBCnodes];
      q_dirNE  = &QQ[dPP0 * numberOfBCnodes];
      q_dirSW  = &QQ[dMM0 * numberOfBCnodes];
      q_dirSE  = &QQ[dPM0 * numberOfBCnodes];
      q_dirNW  = &QQ[dMP0 * numberOfBCnodes];
      q_dirTE  = &QQ[dP0P * numberOfBCnodes];
      q_dirBW  = &QQ[dM0M * numberOfBCnodes];
      q_dirBE  = &QQ[dP0M * numberOfBCnodes];
      q_dirTW  = &QQ[dM0P * numberOfBCnodes];
      q_dirTN  = &QQ[d0PP * numberOfBCnodes];
      q_dirBS  = &QQ[d0MM * numberOfBCnodes];
      q_dirBN  = &QQ[d0PM * numberOfBCnodes];
      q_dirTS  = &QQ[d0MP * numberOfBCnodes];
      q_dirTNE = &QQ[dPPP * numberOfBCnodes];
      q_dirTSW = &QQ[dMMP * numberOfBCnodes];
      q_dirTSE = &QQ[dPMP * numberOfBCnodes];
      q_dirTNW = &QQ[dMPP * numberOfBCnodes];
      q_dirBNE = &QQ[dPPM * numberOfBCnodes];
      q_dirBSW = &QQ[dMMM * numberOfBCnodes];
      q_dirBSE = &QQ[dPMM * numberOfBCnodes];
      q_dirBNW = &QQ[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[dP00 * numberOfBCnodes];
      nx_dirW   = &NormalX[dM00 * numberOfBCnodes];
      nx_dirN   = &NormalX[d0P0 * numberOfBCnodes];
      nx_dirS   = &NormalX[d0M0 * numberOfBCnodes];
      nx_dirT   = &NormalX[d00P * numberOfBCnodes];
      nx_dirB   = &NormalX[d00M * numberOfBCnodes];
      nx_dirNE  = &NormalX[dPP0 * numberOfBCnodes];
      nx_dirSW  = &NormalX[dMM0 * numberOfBCnodes];
      nx_dirSE  = &NormalX[dPM0 * numberOfBCnodes];
      nx_dirNW  = &NormalX[dMP0 * numberOfBCnodes];
      nx_dirTE  = &NormalX[dP0P * numberOfBCnodes];
      nx_dirBW  = &NormalX[dM0M * numberOfBCnodes];
      nx_dirBE  = &NormalX[dP0M * numberOfBCnodes];
      nx_dirTW  = &NormalX[dM0P * numberOfBCnodes];
      nx_dirTN  = &NormalX[d0PP * numberOfBCnodes];
      nx_dirBS  = &NormalX[d0MM * numberOfBCnodes];
      nx_dirBN  = &NormalX[d0PM * numberOfBCnodes];
      nx_dirTS  = &NormalX[d0MP * numberOfBCnodes];
      nx_dirTNE = &NormalX[dPPP * numberOfBCnodes];
      nx_dirTSW = &NormalX[dMMP * numberOfBCnodes];
      nx_dirTSE = &NormalX[dPMP * numberOfBCnodes];
      nx_dirTNW = &NormalX[dMPP * numberOfBCnodes];
      nx_dirBNE = &NormalX[dPPM * numberOfBCnodes];
      nx_dirBSW = &NormalX[dMMM * numberOfBCnodes];
      nx_dirBSE = &NormalX[dPMM * numberOfBCnodes];
      nx_dirBNW = &NormalX[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[dP00 * numberOfBCnodes];
      ny_dirW   = &NormalY[dM00 * numberOfBCnodes];
      ny_dirN   = &NormalY[d0P0 * numberOfBCnodes];
      ny_dirS   = &NormalY[d0M0 * numberOfBCnodes];
      ny_dirT   = &NormalY[d00P * numberOfBCnodes];
      ny_dirB   = &NormalY[d00M * numberOfBCnodes];
      ny_dirNE  = &NormalY[dPP0 * numberOfBCnodes];
      ny_dirSW  = &NormalY[dMM0 * numberOfBCnodes];
      ny_dirSE  = &NormalY[dPM0 * numberOfBCnodes];
      ny_dirNW  = &NormalY[dMP0 * numberOfBCnodes];
      ny_dirTE  = &NormalY[dP0P * numberOfBCnodes];
      ny_dirBW  = &NormalY[dM0M * numberOfBCnodes];
      ny_dirBE  = &NormalY[dP0M * numberOfBCnodes];
      ny_dirTW  = &NormalY[dM0P * numberOfBCnodes];
      ny_dirTN  = &NormalY[d0PP * numberOfBCnodes];
      ny_dirBS  = &NormalY[d0MM * numberOfBCnodes];
      ny_dirBN  = &NormalY[d0PM * numberOfBCnodes];
      ny_dirTS  = &NormalY[d0MP * numberOfBCnodes];
      ny_dirTNE = &NormalY[dPPP * numberOfBCnodes];
      ny_dirTSW = &NormalY[dMMP * numberOfBCnodes];
      ny_dirTSE = &NormalY[dPMP * numberOfBCnodes];
      ny_dirTNW = &NormalY[dMPP * numberOfBCnodes];
      ny_dirBNE = &NormalY[dPPM * numberOfBCnodes];
      ny_dirBSW = &NormalY[dMMM * numberOfBCnodes];
      ny_dirBSE = &NormalY[dPMM * numberOfBCnodes];
      ny_dirBNW = &NormalY[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[dP00 * numberOfBCnodes];
      nz_dirW   = &NormalZ[dM00 * numberOfBCnodes];
      nz_dirN   = &NormalZ[d0P0 * numberOfBCnodes];
      nz_dirS   = &NormalZ[d0M0 * numberOfBCnodes];
      nz_dirT   = &NormalZ[d00P * numberOfBCnodes];
      nz_dirB   = &NormalZ[d00M * numberOfBCnodes];
      nz_dirNE  = &NormalZ[dPP0 * numberOfBCnodes];
      nz_dirSW  = &NormalZ[dMM0 * numberOfBCnodes];
      nz_dirSE  = &NormalZ[dPM0 * numberOfBCnodes];
      nz_dirNW  = &NormalZ[dMP0 * numberOfBCnodes];
      nz_dirTE  = &NormalZ[dP0P * numberOfBCnodes];
      nz_dirBW  = &NormalZ[dM0M * numberOfBCnodes];
      nz_dirBE  = &NormalZ[dP0M * numberOfBCnodes];
      nz_dirTW  = &NormalZ[dM0P * numberOfBCnodes];
      nz_dirTN  = &NormalZ[d0PP * numberOfBCnodes];
      nz_dirBS  = &NormalZ[d0MM * numberOfBCnodes];
      nz_dirBN  = &NormalZ[d0PM * numberOfBCnodes];
      nz_dirTS  = &NormalZ[d0MP * numberOfBCnodes];
      nz_dirTNE = &NormalZ[dPPP * numberOfBCnodes];
      nz_dirTSW = &NormalZ[dMMP * numberOfBCnodes];
      nz_dirTSE = &NormalZ[dPMP * numberOfBCnodes];
      nz_dirTNW = &NormalZ[dMPP * numberOfBCnodes];
      nz_dirBNE = &NormalZ[dPPM * numberOfBCnodes];
      nz_dirBSW = &NormalZ[dMMM * numberOfBCnodes];
      nz_dirBSE = &NormalZ[dPMM * numberOfBCnodes];
      nz_dirBNW = &NormalZ[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[dP00])[ke   ];
      real f_E    = (D.f[dM00])[kw   ];
      real f_S    = (D.f[d0P0])[kn   ];
      real f_N    = (D.f[d0M0])[ks   ];
      real f_B    = (D.f[d00P])[kt   ];
      real f_T    = (D.f[d00M])[kb   ];
      real f_SW   = (D.f[dPP0])[kne  ];
      real f_NE   = (D.f[dMM0])[ksw  ];
      real f_NW   = (D.f[dPM0])[kse  ];
      real f_SE   = (D.f[dMP0])[knw  ];
      real f_BW   = (D.f[dP0P])[kte  ];
      real f_TE   = (D.f[dM0M])[kbw  ];
      real f_TW   = (D.f[dP0M])[kbe  ];
      real f_BE   = (D.f[dM0P])[ktw  ];
      real f_BS   = (D.f[d0PP])[ktn  ];
      real f_TN   = (D.f[d0MM])[kbs  ];
      real f_TS   = (D.f[d0PM])[kbn  ];
      real f_BN   = (D.f[d0MP])[kts  ];
      real f_BSW  = (D.f[dPPP])[ktne ];
      real f_BNE  = (D.f[dMMP])[ktsw ];
      real f_BNW  = (D.f[dPMP])[ktse ];
      real f_BSE  = (D.f[dMPP])[ktnw ];
      real f_TSW  = (D.f[dPPM])[kbne ];
      real f_TNE  = (D.f[dMMM])[kbsw ];
      real f_TNW  = (D.f[dPMM])[kbse ];
      real f_TSE  = (D.f[dMPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dP00] = &DD[dP00 * numberOfLBnodes];
         D.f[dM00] = &DD[dM00 * numberOfLBnodes];
         D.f[d0P0] = &DD[d0P0 * numberOfLBnodes];
         D.f[d0M0] = &DD[d0M0 * numberOfLBnodes];
         D.f[d00P] = &DD[d00P * numberOfLBnodes];
         D.f[d00M] = &DD[d00M * numberOfLBnodes];
         D.f[dPP0] = &DD[dPP0 * numberOfLBnodes];
         D.f[dMM0] = &DD[dMM0 * numberOfLBnodes];
         D.f[dPM0] = &DD[dPM0 * numberOfLBnodes];
         D.f[dMP0] = &DD[dMP0 * numberOfLBnodes];
         D.f[dP0P] = &DD[dP0P * numberOfLBnodes];
         D.f[dM0M] = &DD[dM0M * numberOfLBnodes];
         D.f[dP0M] = &DD[dP0M * numberOfLBnodes];
         D.f[dM0P] = &DD[dM0P * numberOfLBnodes];
         D.f[d0PP] = &DD[d0PP * numberOfLBnodes];
         D.f[d0MM] = &DD[d0MM * numberOfLBnodes];
         D.f[d0PM] = &DD[d0PM * numberOfLBnodes];
         D.f[d0MP] = &DD[d0MP * numberOfLBnodes];
         D.f[d000] = &DD[d000 * numberOfLBnodes];
         D.f[dPPP] = &DD[dPPP * numberOfLBnodes];
         D.f[dMMP] = &DD[dMMP * numberOfLBnodes];
         D.f[dPMP] = &DD[dPMP * numberOfLBnodes];
         D.f[dMPP] = &DD[dMPP * numberOfLBnodes];
         D.f[dPPM] = &DD[dPPM * numberOfLBnodes];
         D.f[dMMM] = &DD[dMMM * numberOfLBnodes];
         D.f[dPMM] = &DD[dPMM * numberOfLBnodes];
         D.f[dMPM] = &DD[dMPM * numberOfLBnodes];
      } 
      else
      {
         D.f[dM00] = &DD[dP00 * numberOfLBnodes];
         D.f[dP00] = &DD[dM00 * numberOfLBnodes];
         D.f[d0M0] = &DD[d0P0 * numberOfLBnodes];
         D.f[d0P0] = &DD[d0M0 * numberOfLBnodes];
         D.f[d00M] = &DD[d00P * numberOfLBnodes];
         D.f[d00P] = &DD[d00M * numberOfLBnodes];
         D.f[dMM0] = &DD[dPP0 * numberOfLBnodes];
         D.f[dPP0] = &DD[dMM0 * numberOfLBnodes];
         D.f[dMP0] = &DD[dPM0 * numberOfLBnodes];
         D.f[dPM0] = &DD[dMP0 * numberOfLBnodes];
         D.f[dM0M] = &DD[dP0P * numberOfLBnodes];
         D.f[dP0P] = &DD[dM0M * numberOfLBnodes];
         D.f[dM0P] = &DD[dP0M * numberOfLBnodes];
         D.f[dP0M] = &DD[dM0P * numberOfLBnodes];
         D.f[d0MM] = &DD[d0PP * numberOfLBnodes];
         D.f[d0PP] = &DD[d0MM * numberOfLBnodes];
         D.f[d0MP] = &DD[d0PM * numberOfLBnodes];
         D.f[d0PM] = &DD[d0MP * numberOfLBnodes];
         D.f[d000] = &DD[d000 * numberOfLBnodes];
         D.f[dPPP] = &DD[dMMM * numberOfLBnodes];
         D.f[dMMP] = &DD[dPPM * numberOfLBnodes];
         D.f[dPMP] = &DD[dMPM * numberOfLBnodes];
         D.f[dMPP] = &DD[dPMM * numberOfLBnodes];
         D.f[dPPM] = &DD[dMMP * numberOfLBnodes];
         D.f[dMMM] = &DD[dPPP * numberOfLBnodes];
         D.f[dPMM] = &DD[dMPP * numberOfLBnodes];
         D.f[dMPM] = &DD[dPMP * numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX = vx1;
      real VeloY = vx2;
      real VeloZ = vx3;
      real fac = c0o1;//0.5;
       real phi = c0o1;
      //real alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real kxyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho) - ((vx1*vx2)));
      real kyzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho) - ((vx2*vx3)));
      real kxzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho) - ((vx1*vx3)));

      real kxxFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx1*vx1)));
      real kyyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx2*vx2)));
      real kzzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx3*vx3)));

      real magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

      //fac = fac * magS / (c1o3 * (one / om1 - c1o2));
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //real *facAst = &QQ[d000 * numberOfBCnodes];

      //fac = fac * alpha + facAst[k] * (one - alpha);
      //facAst[k] = fac;
      //(&QQ[d000 * numberOfBCnodes])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      ////real uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //real phi = expf(magS/0.01f) - one;
      //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //real C = five;
      //real kappa = 0.41f;
      //real phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
      //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //real sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
      //real qMed = sum/count;
      //real phi = fac / (qMed + fac);
      //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real testQ = c2o1;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirE[k]) + fac);
         VeloX *= phi;
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dM00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirW[k]) + fac);
         VeloX *= phi;
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dP00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( ny_dirN[k]) + fac);
         VeloY *= phi;
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[d0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-ny_dirS[k]) + fac);
         VeloY *= phi;
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[d0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nz_dirT[k]) + fac);
         VeloZ *= phi;
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nz_dirB[k]) + fac);
         VeloZ *= phi;
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
         VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirNE[k] + ny_dirNE[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dMM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
         VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirSW[k] - ny_dirSW[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dPP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
         VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirSE[k] - ny_dirSE[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dMP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
         VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirNW[k] + ny_dirNW[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dPM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
         VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirTE[k] + nz_dirTE[k]) + fac);
         VeloX *= phi;
         VeloZ *= phi;
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dM0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
         VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirBW[k] - nz_dirBW[k]) + fac);
         VeloX *= phi;
         VeloZ *= phi;
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dP0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
         VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirBE[k] - nz_dirBE[k]) + fac);
         VeloX *= phi;
         VeloZ *= phi;
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dM0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
         VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirTW[k] + nz_dirTW[k]) + fac);
         VeloX *= phi;
         VeloZ *= phi;
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dP0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
         VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( ny_dirTN[k] + nz_dirTN[k]) + fac);
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
         VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-ny_dirBS[k] - nz_dirBS[k]) + fac);
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
         VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( ny_dirBN[k] - nz_dirBN[k]) + fac);
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
         VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-ny_dirTS[k] + nz_dirTS[k]) + fac);
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
         VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
         VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dMMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
         VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
         VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dPPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
         VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
         VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dMMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
         VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
         VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dPPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
         VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
         VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dMPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
         VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
         VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dPMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
         VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
         VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = (phi > one) ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dMPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
         VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
         VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
         //phi = fac * (one + magS / (c0p0000002+uk) * (one-q));
         //phi = phi > one ? one:phi;
         //phi = fac; //Test
         q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
         phi = fac / (q * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]) + fac);
         VeloX *= phi;
         VeloY *= phi;
         VeloZ *= phi;
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dPMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q) - c1o216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
__global__ void QSlipNormDeviceComp27(
    real* DD, 
    int* k_Q, 
    real* QQ,
    unsigned int  numberOfBCnodes,
    real om1, 
    real* NormalX,
    real* NormalY,
    real* NormalZ,
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dP00] = &DD[dP00 * numberOfLBnodes];
      D.f[dM00] = &DD[dM00 * numberOfLBnodes];
      D.f[d0P0] = &DD[d0P0 * numberOfLBnodes];
      D.f[d0M0] = &DD[d0M0 * numberOfLBnodes];
      D.f[d00P] = &DD[d00P * numberOfLBnodes];
      D.f[d00M] = &DD[d00M * numberOfLBnodes];
      D.f[dPP0] = &DD[dPP0 * numberOfLBnodes];
      D.f[dMM0] = &DD[dMM0 * numberOfLBnodes];
      D.f[dPM0] = &DD[dPM0 * numberOfLBnodes];
      D.f[dMP0] = &DD[dMP0 * numberOfLBnodes];
      D.f[dP0P] = &DD[dP0P * numberOfLBnodes];
      D.f[dM0M] = &DD[dM0M * numberOfLBnodes];
      D.f[dP0M] = &DD[dP0M * numberOfLBnodes];
      D.f[dM0P] = &DD[dM0P * numberOfLBnodes];
      D.f[d0PP] = &DD[d0PP * numberOfLBnodes];
      D.f[d0MM] = &DD[d0MM * numberOfLBnodes];
      D.f[d0PM] = &DD[d0PM * numberOfLBnodes];
      D.f[d0MP] = &DD[d0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[dPPP] = &DD[dPPP * numberOfLBnodes];
      D.f[dMMP] = &DD[dMMP * numberOfLBnodes];
      D.f[dPMP] = &DD[dPMP * numberOfLBnodes];
      D.f[dMPP] = &DD[dMPP * numberOfLBnodes];
      D.f[dPPM] = &DD[dPPM * numberOfLBnodes];
      D.f[dMMM] = &DD[dMMM * numberOfLBnodes];
      D.f[dPMM] = &DD[dPMM * numberOfLBnodes];
      D.f[dMPM] = &DD[dMPM * numberOfLBnodes];
   } 
   else
   {
      D.f[dM00] = &DD[dP00 * numberOfLBnodes];
      D.f[dP00] = &DD[dM00 * numberOfLBnodes];
      D.f[d0M0] = &DD[d0P0 * numberOfLBnodes];
      D.f[d0P0] = &DD[d0M0 * numberOfLBnodes];
      D.f[d00M] = &DD[d00P * numberOfLBnodes];
      D.f[d00P] = &DD[d00M * numberOfLBnodes];
      D.f[dMM0] = &DD[dPP0 * numberOfLBnodes];
      D.f[dPP0] = &DD[dMM0 * numberOfLBnodes];
      D.f[dMP0] = &DD[dPM0 * numberOfLBnodes];
      D.f[dPM0] = &DD[dMP0 * numberOfLBnodes];
      D.f[dM0M] = &DD[dP0P * numberOfLBnodes];
      D.f[dP0P] = &DD[dM0M * numberOfLBnodes];
      D.f[dM0P] = &DD[dP0M * numberOfLBnodes];
      D.f[dP0M] = &DD[dM0P * numberOfLBnodes];
      D.f[d0MM] = &DD[d0PP * numberOfLBnodes];
      D.f[d0PP] = &DD[d0MM * numberOfLBnodes];
      D.f[d0MP] = &DD[d0PM * numberOfLBnodes];
      D.f[d0PM] = &DD[d0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[dPPP] = &DD[dMMM * numberOfLBnodes];
      D.f[dMMP] = &DD[dPPM * numberOfLBnodes];
      D.f[dPMP] = &DD[dMPM * numberOfLBnodes];
      D.f[dMPP] = &DD[dPMM * numberOfLBnodes];
      D.f[dPPM] = &DD[dMMP * numberOfLBnodes];
      D.f[dMMM] = &DD[dPPP * numberOfLBnodes];
      D.f[dPMM] = &DD[dMPP * numberOfLBnodes];
      D.f[dMPM] = &DD[dPMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dP00 * numberOfBCnodes];
      q_dirW   = &QQ[dM00 * numberOfBCnodes];
      q_dirN   = &QQ[d0P0 * numberOfBCnodes];
      q_dirS   = &QQ[d0M0 * numberOfBCnodes];
      q_dirT   = &QQ[d00P * numberOfBCnodes];
      q_dirB   = &QQ[d00M * numberOfBCnodes];
      q_dirNE  = &QQ[dPP0 * numberOfBCnodes];
      q_dirSW  = &QQ[dMM0 * numberOfBCnodes];
      q_dirSE  = &QQ[dPM0 * numberOfBCnodes];
      q_dirNW  = &QQ[dMP0 * numberOfBCnodes];
      q_dirTE  = &QQ[dP0P * numberOfBCnodes];
      q_dirBW  = &QQ[dM0M * numberOfBCnodes];
      q_dirBE  = &QQ[dP0M * numberOfBCnodes];
      q_dirTW  = &QQ[dM0P * numberOfBCnodes];
      q_dirTN  = &QQ[d0PP * numberOfBCnodes];
      q_dirBS  = &QQ[d0MM * numberOfBCnodes];
      q_dirBN  = &QQ[d0PM * numberOfBCnodes];
      q_dirTS  = &QQ[d0MP * numberOfBCnodes];
      q_dirTNE = &QQ[dPPP * numberOfBCnodes];
      q_dirTSW = &QQ[dMMP * numberOfBCnodes];
      q_dirTSE = &QQ[dPMP * numberOfBCnodes];
      q_dirTNW = &QQ[dMPP * numberOfBCnodes];
      q_dirBNE = &QQ[dPPM * numberOfBCnodes];
      q_dirBSW = &QQ[dMMM * numberOfBCnodes];
      q_dirBSE = &QQ[dPMM * numberOfBCnodes];
      q_dirBNW = &QQ[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[dP00 * numberOfBCnodes];
      nx_dirW   = &NormalX[dM00 * numberOfBCnodes];
      nx_dirN   = &NormalX[d0P0 * numberOfBCnodes];
      nx_dirS   = &NormalX[d0M0 * numberOfBCnodes];
      nx_dirT   = &NormalX[d00P * numberOfBCnodes];
      nx_dirB   = &NormalX[d00M * numberOfBCnodes];
      nx_dirNE  = &NormalX[dPP0 * numberOfBCnodes];
      nx_dirSW  = &NormalX[dMM0 * numberOfBCnodes];
      nx_dirSE  = &NormalX[dPM0 * numberOfBCnodes];
      nx_dirNW  = &NormalX[dMP0 * numberOfBCnodes];
      nx_dirTE  = &NormalX[dP0P * numberOfBCnodes];
      nx_dirBW  = &NormalX[dM0M * numberOfBCnodes];
      nx_dirBE  = &NormalX[dP0M * numberOfBCnodes];
      nx_dirTW  = &NormalX[dM0P * numberOfBCnodes];
      nx_dirTN  = &NormalX[d0PP * numberOfBCnodes];
      nx_dirBS  = &NormalX[d0MM * numberOfBCnodes];
      nx_dirBN  = &NormalX[d0PM * numberOfBCnodes];
      nx_dirTS  = &NormalX[d0MP * numberOfBCnodes];
      nx_dirTNE = &NormalX[dPPP * numberOfBCnodes];
      nx_dirTSW = &NormalX[dMMP * numberOfBCnodes];
      nx_dirTSE = &NormalX[dPMP * numberOfBCnodes];
      nx_dirTNW = &NormalX[dMPP * numberOfBCnodes];
      nx_dirBNE = &NormalX[dPPM * numberOfBCnodes];
      nx_dirBSW = &NormalX[dMMM * numberOfBCnodes];
      nx_dirBSE = &NormalX[dPMM * numberOfBCnodes];
      nx_dirBNW = &NormalX[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[dP00 * numberOfBCnodes];
      ny_dirW   = &NormalY[dM00 * numberOfBCnodes];
      ny_dirN   = &NormalY[d0P0 * numberOfBCnodes];
      ny_dirS   = &NormalY[d0M0 * numberOfBCnodes];
      ny_dirT   = &NormalY[d00P * numberOfBCnodes];
      ny_dirB   = &NormalY[d00M * numberOfBCnodes];
      ny_dirNE  = &NormalY[dPP0 * numberOfBCnodes];
      ny_dirSW  = &NormalY[dMM0 * numberOfBCnodes];
      ny_dirSE  = &NormalY[dPM0 * numberOfBCnodes];
      ny_dirNW  = &NormalY[dMP0 * numberOfBCnodes];
      ny_dirTE  = &NormalY[dP0P * numberOfBCnodes];
      ny_dirBW  = &NormalY[dM0M * numberOfBCnodes];
      ny_dirBE  = &NormalY[dP0M * numberOfBCnodes];
      ny_dirTW  = &NormalY[dM0P * numberOfBCnodes];
      ny_dirTN  = &NormalY[d0PP * numberOfBCnodes];
      ny_dirBS  = &NormalY[d0MM * numberOfBCnodes];
      ny_dirBN  = &NormalY[d0PM * numberOfBCnodes];
      ny_dirTS  = &NormalY[d0MP * numberOfBCnodes];
      ny_dirTNE = &NormalY[dPPP * numberOfBCnodes];
      ny_dirTSW = &NormalY[dMMP * numberOfBCnodes];
      ny_dirTSE = &NormalY[dPMP * numberOfBCnodes];
      ny_dirTNW = &NormalY[dMPP * numberOfBCnodes];
      ny_dirBNE = &NormalY[dPPM * numberOfBCnodes];
      ny_dirBSW = &NormalY[dMMM * numberOfBCnodes];
      ny_dirBSE = &NormalY[dPMM * numberOfBCnodes];
      ny_dirBNW = &NormalY[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      real *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[dP00 * numberOfBCnodes];
      nz_dirW   = &NormalZ[dM00 * numberOfBCnodes];
      nz_dirN   = &NormalZ[d0P0 * numberOfBCnodes];
      nz_dirS   = &NormalZ[d0M0 * numberOfBCnodes];
      nz_dirT   = &NormalZ[d00P * numberOfBCnodes];
      nz_dirB   = &NormalZ[d00M * numberOfBCnodes];
      nz_dirNE  = &NormalZ[dPP0 * numberOfBCnodes];
      nz_dirSW  = &NormalZ[dMM0 * numberOfBCnodes];
      nz_dirSE  = &NormalZ[dPM0 * numberOfBCnodes];
      nz_dirNW  = &NormalZ[dMP0 * numberOfBCnodes];
      nz_dirTE  = &NormalZ[dP0P * numberOfBCnodes];
      nz_dirBW  = &NormalZ[dM0M * numberOfBCnodes];
      nz_dirBE  = &NormalZ[dP0M * numberOfBCnodes];
      nz_dirTW  = &NormalZ[dM0P * numberOfBCnodes];
      nz_dirTN  = &NormalZ[d0PP * numberOfBCnodes];
      nz_dirBS  = &NormalZ[d0MM * numberOfBCnodes];
      nz_dirBN  = &NormalZ[d0PM * numberOfBCnodes];
      nz_dirTS  = &NormalZ[d0MP * numberOfBCnodes];
      nz_dirTNE = &NormalZ[dPPP * numberOfBCnodes];
      nz_dirTSW = &NormalZ[dMMP * numberOfBCnodes];
      nz_dirTSE = &NormalZ[dPMP * numberOfBCnodes];
      nz_dirTNW = &NormalZ[dMPP * numberOfBCnodes];
      nz_dirBNE = &NormalZ[dPPM * numberOfBCnodes];
      nz_dirBSW = &NormalZ[dMMM * numberOfBCnodes];
      nz_dirBSE = &NormalZ[dPMM * numberOfBCnodes];
      nz_dirBNW = &NormalZ[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_W    = (D.f[dP00])[ke   ];
      real f_E    = (D.f[dM00])[kw   ];
      real f_S    = (D.f[d0P0])[kn   ];
      real f_N    = (D.f[d0M0])[ks   ];
      real f_B    = (D.f[d00P])[kt   ];
      real f_T    = (D.f[d00M])[kb   ];
      real f_SW   = (D.f[dPP0])[kne  ];
      real f_NE   = (D.f[dMM0])[ksw  ];
      real f_NW   = (D.f[dPM0])[kse  ];
      real f_SE   = (D.f[dMP0])[knw  ];
      real f_BW   = (D.f[dP0P])[kte  ];
      real f_TE   = (D.f[dM0M])[kbw  ];
      real f_TW   = (D.f[dP0M])[kbe  ];
      real f_BE   = (D.f[dM0P])[ktw  ];
      real f_BS   = (D.f[d0PP])[ktn  ];
      real f_TN   = (D.f[d0MM])[kbs  ];
      real f_TS   = (D.f[d0PM])[kbn  ];
      real f_BN   = (D.f[d0MP])[kts  ];
      real f_BSW  = (D.f[dPPP])[ktne ];
      real f_BNE  = (D.f[dMMP])[ktsw ];
      real f_BNW  = (D.f[dPMP])[ktse ];
      real f_BSE  = (D.f[dMPP])[ktnw ];
      real f_TSW  = (D.f[dPPM])[kbne ];
      real f_TNE  = (D.f[dMMM])[kbsw ];
      real f_TNW  = (D.f[dPMM])[kbse ];
      real f_TSE  = (D.f[dMPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dP00] = &DD[dP00 * numberOfLBnodes];
         D.f[dM00] = &DD[dM00 * numberOfLBnodes];
         D.f[d0P0] = &DD[d0P0 * numberOfLBnodes];
         D.f[d0M0] = &DD[d0M0 * numberOfLBnodes];
         D.f[d00P] = &DD[d00P * numberOfLBnodes];
         D.f[d00M] = &DD[d00M * numberOfLBnodes];
         D.f[dPP0] = &DD[dPP0 * numberOfLBnodes];
         D.f[dMM0] = &DD[dMM0 * numberOfLBnodes];
         D.f[dPM0] = &DD[dPM0 * numberOfLBnodes];
         D.f[dMP0] = &DD[dMP0 * numberOfLBnodes];
         D.f[dP0P] = &DD[dP0P * numberOfLBnodes];
         D.f[dM0M] = &DD[dM0M * numberOfLBnodes];
         D.f[dP0M] = &DD[dP0M * numberOfLBnodes];
         D.f[dM0P] = &DD[dM0P * numberOfLBnodes];
         D.f[d0PP] = &DD[d0PP * numberOfLBnodes];
         D.f[d0MM] = &DD[d0MM * numberOfLBnodes];
         D.f[d0PM] = &DD[d0PM * numberOfLBnodes];
         D.f[d0MP] = &DD[d0MP * numberOfLBnodes];
         D.f[d000] = &DD[d000 * numberOfLBnodes];
         D.f[dPPP] = &DD[dPPP * numberOfLBnodes];
         D.f[dMMP] = &DD[dMMP * numberOfLBnodes];
         D.f[dPMP] = &DD[dPMP * numberOfLBnodes];
         D.f[dMPP] = &DD[dMPP * numberOfLBnodes];
         D.f[dPPM] = &DD[dPPM * numberOfLBnodes];
         D.f[dMMM] = &DD[dMMM * numberOfLBnodes];
         D.f[dPMM] = &DD[dPMM * numberOfLBnodes];
         D.f[dMPM] = &DD[dMPM * numberOfLBnodes];
      } 
      else
      {
         D.f[dM00] = &DD[dP00 * numberOfLBnodes];
         D.f[dP00] = &DD[dM00 * numberOfLBnodes];
         D.f[d0M0] = &DD[d0P0 * numberOfLBnodes];
         D.f[d0P0] = &DD[d0M0 * numberOfLBnodes];
         D.f[d00M] = &DD[d00P * numberOfLBnodes];
         D.f[d00P] = &DD[d00M * numberOfLBnodes];
         D.f[dMM0] = &DD[dPP0 * numberOfLBnodes];
         D.f[dPP0] = &DD[dMM0 * numberOfLBnodes];
         D.f[dMP0] = &DD[dPM0 * numberOfLBnodes];
         D.f[dPM0] = &DD[dMP0 * numberOfLBnodes];
         D.f[dM0M] = &DD[dP0P * numberOfLBnodes];
         D.f[dP0P] = &DD[dM0M * numberOfLBnodes];
         D.f[dM0P] = &DD[dP0M * numberOfLBnodes];
         D.f[dP0M] = &DD[dM0P * numberOfLBnodes];
         D.f[d0MM] = &DD[d0PP * numberOfLBnodes];
         D.f[d0PP] = &DD[d0MM * numberOfLBnodes];
         D.f[d0MP] = &DD[d0PM * numberOfLBnodes];
         D.f[d0PM] = &DD[d0MP * numberOfLBnodes];
         D.f[d000] = &DD[d000 * numberOfLBnodes];
         D.f[dPPP] = &DD[dMMM * numberOfLBnodes];
         D.f[dMMP] = &DD[dPPM * numberOfLBnodes];
         D.f[dPMP] = &DD[dMPM * numberOfLBnodes];
         D.f[dMPP] = &DD[dPMM * numberOfLBnodes];
         D.f[dPPM] = &DD[dMMP * numberOfLBnodes];
         D.f[dMMM] = &DD[dPPP * numberOfLBnodes];
         D.f[dPMM] = &DD[dMPP * numberOfLBnodes];
         D.f[dMPM] = &DD[dPMP * numberOfLBnodes];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX = vx1;
      real VeloY = vx2;
      real VeloZ = vx3;
      real fac = c1o100;//0.5;
       //real phi = c0o1;
      real alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real kxyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (c1o1 + drho) - ((vx1*vx2)));
      real kyzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (c1o1 + drho) - ((vx2*vx3)));
      real kxzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (c1o1 + drho) - ((vx1*vx3)));

      real kxxFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx1*vx1)));
      real kyyFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx2*vx2)));
      real kzzFromfcNEQ = -(c3o1 * om1 / (c1o1-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (c1o1 + drho) - ((c1o3*drho + vx3*vx3)));

      real magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

      fac = fac * magS / (c1o3 * (c1o1 / om1 - c1o2));
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real *facAst = &QQ[d000 * numberOfBCnodes];

      fac = fac * alpha + facAst[k] * (c1o1 - alpha);
      facAst[k] = fac;
      //(&QQ[d000 * numberOfBCnodes])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      ////real uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //real phi = expf(magS/0.01f) - one;
      //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //real C = five;
      //real kappa = 0.41f;
      //real phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
      //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //real sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
      //real qMed = sum/count;
      //real phi = fac / (qMed + fac);
      //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real sliplength = 0.9f;//c1o2;
      real qSlip = c0o1;
      real un = c0o1;
      real ut = c0o1;
      real tangential = c0o1;
      //real cSmallSingle = c0p0000002;

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
         un = fabs((vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k]);
         ut = fabs(VeloX);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c2o27* (drho/*+three*( vx1        )*/+c9o2*( vx1        )*( vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dM00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
         un = fabs(-(vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k]);
         ut = fabs(-VeloX);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c2o27* (drho/*+three*(-vx1        )*/+c9o2*(-vx1        )*(-vx1        ) * (c1o1 + drho)-cu_sq); 
         (D.f[dP00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
         un = fabs( (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k]);
         ut = fabs( VeloY);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( ny_dirN[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c2o27* (drho/*+three*(    vx2     )*/+c9o2*(     vx2    )*(     vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[d0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
         un = fabs(-(vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k]);
         ut = fabs(-VeloY);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-ny_dirS[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c2o27* (drho/*+three*(   -vx2     )*/+c9o2*(    -vx2    )*(    -vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[d0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
         un = fabs( (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k]);
         ut = fabs( VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nz_dirT[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c2o27* (drho/*+three*(         vx3)*/+c9o2*(         vx3)*(         vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
         un = fabs(-(vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k]);
         ut = fabs(-VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nz_dirB[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c2o27* (drho/*+three*(        -vx3)*/+c9o2*(        -vx3)*(        -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T))/(c1o1+q) - c2o27 * drho;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
         VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
         un = fabs( (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k] + (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k]);
         ut = fabs( VeloX + VeloY);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirNE[k]+ny_dirNE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*( vx1+vx2    )*/+c9o2*( vx1+vx2    )*( vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dMM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
         VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
         un = fabs(-(vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k] - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k]);
         ut = fabs(-VeloX - VeloY);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirSW[k]-ny_dirSW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1-vx2    )*/+c9o2*(-vx1-vx2    )*(-vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dPP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
         VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
         un = fabs( (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k] - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k]);
         ut = fabs( VeloX - VeloY);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirSE[k]-ny_dirSE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*( vx1-vx2    )*/+c9o2*( vx1-vx2    )*( vx1-vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dMP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
         VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
         un = fabs(-(vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k] + (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k]);
         ut = fabs(-VeloX + VeloY);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirNW[k]+ny_dirNW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1+vx2    )*/+c9o2*(-vx1+vx2    )*(-vx1+vx2    ) * (c1o1 + drho)-cu_sq); 
         (D.f[dPM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
         VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
         un = fabs( (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k] + (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k]);
         ut = fabs( VeloX + VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirTE[k]+nz_dirTE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*( vx1    +vx3)*/+c9o2*( vx1    +vx3)*( vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dM0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
         VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
         un = fabs(-(vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k] - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k]);
         ut = fabs(-VeloX - VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirBW[k]-nz_dirBW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1    -vx3)*/+c9o2*(-vx1    -vx3)*(-vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dP0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
         VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
         un = fabs( (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k] - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k]);
         ut = fabs( VeloX - VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirBE[k]-nz_dirBE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*( vx1    -vx3)*/+c9o2*( vx1    -vx3)*( vx1    -vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dM0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
         VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
         un = fabs(-(vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k] + (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k]);
         ut = fabs(-VeloX + VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirTW[k]+nz_dirTW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*(-vx1    +vx3)*/+c9o2*(-vx1    +vx3)*(-vx1    +vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dP0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
         VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
         un = fabs( (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k] + (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k]);
         ut = fabs( VeloY + VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( ny_dirTN[k]+nz_dirTN[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*(     vx2+vx3)*/+c9o2*(     vx2+vx3)*(     vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
         VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
         un = fabs(-(vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k] - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k]);
         ut = fabs(-VeloY - VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-ny_dirBS[k]-nz_dirBS[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*(    -vx2-vx3)*/+c9o2*(    -vx2-vx3)*(    -vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
         VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
         un = fabs( (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k] - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k]);
         ut = fabs( VeloY - VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( ny_dirBN[k]-nz_dirBN[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*(     vx2-vx3)*/+c9o2*(     vx2-vx3)*(     vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
         VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
         un = fabs(-(vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k] + (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k]);
         ut = fabs(-VeloY + VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-ny_dirTS[k]+nz_dirTS[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o54* (drho/*+three*(    -vx2+vx3)*/+c9o2*(    -vx2+vx3)*(    -vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[d0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN))/(c1o1+q) - c1o54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
         VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
         VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
         un = fabs( (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k] 
                   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k] 
                   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k]);
         ut = fabs( VeloX + VeloY + VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o216*(drho/*+three*( vx1+vx2+vx3)*/+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dMMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
         VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
         VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
         un = fabs(-(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k] 
                   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k] 
                   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k]);
         ut = fabs(-VeloX - VeloY - VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1-vx2-vx3)*/+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dPPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
         VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
         VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
         un = fabs( (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k] 
                   +(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k] 
                   -(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k]);
         ut = fabs( VeloX + VeloY - VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o216*(drho/*+three*( vx1+vx2-vx3)*/+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dMMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
         VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
         VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
         un = fabs(-(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k] 
                   -(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k] 
                   +(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k]);
         ut = fabs(-VeloX - VeloY + VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1-vx2+vx3)*/+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dPPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
         VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
         VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
         un = fabs(+(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k] 
                   -(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k] 
                   +(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k]);
         ut = fabs(+VeloX - VeloY + VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o216*(drho/*+three*( vx1-vx2+vx3)*/+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dMPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
         VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
         VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
         un = fabs(-(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k] 
                   +(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k] 
                   -(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k]);
         ut = fabs(-VeloX + VeloY - VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1+vx2-vx3)*/+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dPMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
         VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
         VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
         un = fabs( (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k] 
                   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k] 
                   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k]);
         ut = fabs( VeloX - VeloY - VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o216*(drho/*+three*( vx1-vx2-vx3)*/+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dMPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW))/(c1o1+q) - c1o216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
         VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
         VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
         un = fabs(-(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k] 
                   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k] 
                   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k]);
         ut = fabs(-VeloX + VeloY + VeloZ);
         tangential = ut / (ut + un + cSmallSingle);
         qSlip = sliplength * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]);        //sliplength * e_i * n_i
         //qSlip = (qSlip < zero) ? zero:qSlip;
         //tangential = (tangential > one) ? one:tangential;
         q = (q + qSlip)/(c1o1 + qSlip * (c1o1 - tangential) / (cSmallSingle + q));
         feq=c1o216*(drho/*+three*(-vx1+vx2+vx3)*/+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (c1o1 + drho)-cu_sq); 
         (D.f[dPMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE))/(c1o1+q) - c1o216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
