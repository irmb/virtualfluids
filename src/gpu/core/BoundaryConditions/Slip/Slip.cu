//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \author Martin Schoenherr
//=======================================================================================
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "Calculation/Calculation.h"
#include <cuda_helper/CudaGrid.h>

#include "BoundaryConditions/Slip/Slip_Device.cuh"
#include "Parameter/Parameter.h"

void SlipCompressible(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
    dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads, boundaryCondition->numberOfBCnodes);
    dim3 threads(parameterDevice->numberofthreads, 1, 1 );

    SlipCompressible_Device<<< grid, threads >>> (
        parameterDevice->distributions.f[0],
        boundaryCondition->k,
        boundaryCondition->q27[0],
        boundaryCondition->numberOfBCnodes,
        parameterDevice->omega,
        parameterDevice->neighborX,
        parameterDevice->neighborY,
        parameterDevice->neighborZ,
        parameterDevice->numberOfNodes,
        parameterDevice->isEvenTimestep);
    getLastCudaError("QSlipDeviceComp27 execution failed");
}


void SlipTurbulentViscosityCompressible(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
    dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads, boundaryCondition->numberOfBCnodes);
    dim3 threads(parameterDevice->numberofthreads, 1, 1 );

    SlipTurbulentViscosityCompressible_Device<<< grid, threads >>> (
        parameterDevice->distributions.f[0],
        boundaryCondition->k,
        boundaryCondition->q27[0],
        boundaryCondition->numberOfBCnodes,
        parameterDevice->omega,
        parameterDevice->neighborX,
        parameterDevice->neighborY,
        parameterDevice->neighborZ,
        parameterDevice->turbViscosity,
        parameterDevice->numberOfNodes,
        parameterDevice->isEvenTimestep);
    getLastCudaError("SlipTurbulentViscosityCompressible_Device execution failed");
}
