#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
//  for more details.
//
//  SPDX-License-Identifier: GPL-3.0-or-later
//  SPDX-FileCopyrightText: Copyright © VirtualFluids Project contributors, see AUTHORS.md in root folder
//
//! \addtogroup gpu_BoundaryConditions BoundaryConditions
//! \ingroup gpu_core core
//! \{
//! \author Henrik Asmuth, Martin Schönherr
//! \brief Kernel for StressBC using the iMEM approach
//!
//! kernel prescribe a wall shear stress using the iMEM apprach (see, Asmuth et. al (2021), https://doi.org/10.1063/5.0065701)
//! StressCompressible_Device couples the iMEM to the single-node interpolated bounce-back.
//! StressBounceBackCompressible_Device couples the iMEM to a simple bounce-back.
//! Note, that the iMEM function is currently only implemented for straight walls with z-normal and q=0.5.
//! Other wall models could be implemented in the iMEM by replacing the formulations from Monin-Obukhov similarity theory (MOST)
//! with other formulations, e.g., for smooth walls.
//! iMEM so far most extensively tested with StressBounceBackCompressible_Device, but StressCompressible_Device also seems to be stable and working.
//=======================================================================================

#include "BoundaryConditions/Stress/iMEM.cuh"

using namespace vf::basics::constant;
using namespace vf::lbm::dir;
using namespace vf::gpu;

__global__ void StressBounceBackPressureCompressible_Device(
    real* DD,
    int* k_Q,
    int* k_N,
    real* QQ,
    unsigned int  numberOfBCnodes,
    real* vx,
    real* vy,
    real* vz,
    real* normalX,
    real* normalY,
    real* normalZ,
    real* vx_el,
    real* vy_el,
    real* vz_el,
    real* vx_w_mean,
    real* vy_w_mean,
    real* vz_w_mean,
    int* samplingOffset,
    real* z0,
    bool  hasWallModelMonitor,
    real* u_star_monitor,
    real* Fx_monitor,
    real* Fy_monitor,
    real* Fz_monitor,
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes,
    bool isEvenTimestep)
{
   Distributions27 D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, isEvenTimestep);

   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index
   const unsigned  y = blockIdx.x;   // Globaler y-Index
   const unsigned  z = blockIdx.y;   // Globaler z-Index

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k< numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB,
         *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
         *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
         *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
         *q_dirBSE, *q_dirBNW;
      q_dirE   = &QQ[dP00 * numberOfBCnodes];
      q_dirW   = &QQ[dM00 * numberOfBCnodes];
      q_dirN   = &QQ[d0P0 * numberOfBCnodes];
      q_dirS   = &QQ[d0M0 * numberOfBCnodes];
      q_dirT   = &QQ[d00P * numberOfBCnodes];
      q_dirB   = &QQ[d00M * numberOfBCnodes];
      q_dirNE  = &QQ[dPP0 * numberOfBCnodes];
      q_dirSW  = &QQ[dMM0 * numberOfBCnodes];
      q_dirSE  = &QQ[dPM0 * numberOfBCnodes];
      q_dirNW  = &QQ[dMP0 * numberOfBCnodes];
      q_dirTE  = &QQ[dP0P * numberOfBCnodes];
      q_dirBW  = &QQ[dM0M * numberOfBCnodes];
      q_dirBE  = &QQ[dP0M * numberOfBCnodes];
      q_dirTW  = &QQ[dM0P * numberOfBCnodes];
      q_dirTN  = &QQ[d0PP * numberOfBCnodes];
      q_dirBS  = &QQ[d0MM * numberOfBCnodes];
      q_dirBN  = &QQ[d0PM * numberOfBCnodes];
      q_dirTS  = &QQ[d0MP * numberOfBCnodes];
      q_dirTNE = &QQ[dPPP * numberOfBCnodes];
      q_dirTSW = &QQ[dMMP * numberOfBCnodes];
      q_dirTSE = &QQ[dPMP * numberOfBCnodes];
      q_dirTNW = &QQ[dMPP * numberOfBCnodes];
      q_dirBNE = &QQ[dPPM * numberOfBCnodes];
      q_dirBSW = &QQ[dMMM * numberOfBCnodes];
      q_dirBSE = &QQ[dPMM * numberOfBCnodes];
      q_dirBNW = &QQ[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];

      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dP00])[ke   ];
      f_E    = (D.f[dM00])[kw   ];
      f_S    = (D.f[d0P0])[kn   ];
      f_N    = (D.f[d0M0])[ks   ];
      f_B    = (D.f[d00P])[kt   ];
      f_T    = (D.f[d00M])[kb   ];
      f_SW   = (D.f[dPP0])[kne  ];
      f_NE   = (D.f[dMM0])[ksw  ];
      f_NW   = (D.f[dPM0])[kse  ];
      f_SE   = (D.f[dMP0])[knw  ];
      f_BW   = (D.f[dP0P])[kte  ];
      f_TE   = (D.f[dM0M])[kbw  ];
      f_TW   = (D.f[dP0M])[kbe  ];
      f_BE   = (D.f[dM0P])[ktw  ];
      f_BS   = (D.f[d0PP])[ktn  ];
      f_TN   = (D.f[d0MM])[kbs  ];
      f_TS   = (D.f[d0PM])[kbn  ];
      f_BN   = (D.f[d0MP])[kts  ];
      f_BSW  = (D.f[dPPP])[ktne ];
      f_BNE  = (D.f[dMMP])[ktsw ];
      f_BNW  = (D.f[dPMP])[ktse ];
      f_BSE  = (D.f[dMPP])[ktnw ];
      f_TSW  = (D.f[dPPM])[kbne ];
      f_TNE  = (D.f[dMMM])[kbsw ];
      f_TNW  = (D.f[dPMM])[kbse ];
      f_TSE  = (D.f[dMPM])[kbnw ];

      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW +
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]);

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (c1o1 + drho);


      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (c1o1 + drho);

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (c1o1 + drho);

      //////////////////////////////////////////////////////////////////////////
      D = vf::gpu::getDistributionReferences27(DD, numberOfLBnodes, !isEvenTimestep);

      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real f_E_in,  f_W_in,  f_N_in,  f_S_in,  f_T_in,  f_B_in,   f_NE_in,  f_SW_in,  f_SE_in,  f_NW_in,  f_TE_in,  f_BW_in,  f_BE_in,
         f_TW_in, f_TN_in, f_BS_in, f_BN_in, f_TS_in, f_TNE_in, f_TSW_in, f_TSE_in, f_TNW_in, f_BNE_in, f_BSW_in, f_BSE_in, f_BNW_in;

      // momentum exchanged with wall at rest
      real wallMomentumX = 0.0, wallMomentumY = 0.0, wallMomentumZ = 0.0;

      real q;
      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_W_in=f_E - c2o27 * drho;
         wallMomentumX += f_E+f_W_in;
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_E_in=f_W - c2o27 * drho;
          wallMomentumX -= f_W+f_E_in;
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_S_in=f_N - c2o27 * drho;
         wallMomentumY += f_N+f_S_in;
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_N_in=f_S - c2o27 * drho;
         wallMomentumY -= f_S+f_N_in;
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_B_in=f_T - c2o27 * drho;
         wallMomentumZ += f_T+f_B_in;
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_T_in=f_B - c2o27 * drho;
         wallMomentumZ -= f_B+f_T_in;
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_SW_in=f_NE - c1o54 * drho;
         wallMomentumX += f_NE+f_SW_in;
         wallMomentumY += f_NE+f_SW_in;
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_NE_in=f_SW - c1o54 * drho;
         wallMomentumX -= f_SW+f_NE_in;
         wallMomentumY -= f_SW+f_NE_in;
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_NW_in=f_SE - c1o54 * drho;
         wallMomentumX += f_SE+f_NW_in;
         wallMomentumY -= f_SE+f_NW_in;
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_SE_in=f_NW - c1o54 * drho;
         wallMomentumX -= f_NW+f_SE_in;
         wallMomentumY += f_NW+f_SE_in;
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BW_in=f_TE - c1o54 * drho;
         wallMomentumX += f_TE+f_BW_in;
         wallMomentumZ += f_TE+f_BW_in;
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TE_in=f_BW - c1o54 * drho;
         wallMomentumX -= f_BW+f_TE_in;
         wallMomentumZ -= f_BW+f_TE_in;
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TW_in=f_BE - c1o54 * drho;
         wallMomentumX += f_BE+f_TW_in;
         wallMomentumZ -= f_BE+f_TW_in;
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BE_in=f_TW - c1o54 * drho;
         wallMomentumX -= f_TW+f_BE_in;
         wallMomentumZ += f_TW+f_BE_in;
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BS_in=f_TN - c1o54 * drho;
         wallMomentumY += f_TN+f_BS_in;
         wallMomentumZ += f_TN+f_BS_in;
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TN_in=f_BS - c1o54 * drho;
         wallMomentumY -= f_BS+f_TN_in;
         wallMomentumZ -= f_BS+f_TN_in;
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TS_in=f_BN - c1o54 * drho;
         wallMomentumY += f_BN+f_TS_in;
         wallMomentumZ -= f_BN+f_TS_in;
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BN_in=f_TS - c1o54 * drho;
         wallMomentumY -= f_TS+f_BN_in;
         wallMomentumZ += f_TS+f_BN_in;
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BSW_in=f_TNE - c1o216 * drho;
         wallMomentumX += f_TNE+f_BSW_in;
         wallMomentumY += f_TNE+f_BSW_in;
         wallMomentumZ += f_TNE+f_BSW_in;
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TNE_in=f_BSW - c1o216 * drho;
         wallMomentumX -= f_BSW+f_TNE_in;
         wallMomentumY -= f_BSW+f_TNE_in;
         wallMomentumZ -= f_BSW+f_TNE_in;
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TSW_in=f_BNE - c1o216 * drho;
         wallMomentumX += f_BNE+f_TSW_in;
         wallMomentumY += f_BNE+f_TSW_in;
         wallMomentumZ -= f_BNE+f_TSW_in;
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BNE_in=f_TSW - c1o216 * drho;
         wallMomentumX -= f_TSW+f_BNE_in;
         wallMomentumY -= f_TSW+f_BNE_in;
         wallMomentumZ += f_TSW+f_BNE_in;
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BNW_in=f_TSE - c1o216 * drho;
         wallMomentumX += f_TSE+f_BNW_in;
         wallMomentumY -= f_TSE+f_BNW_in;
         wallMomentumZ += f_TSE+f_BNW_in;
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TSE_in=f_BNW - c1o216 * drho;
         wallMomentumX -= f_BNW+f_TSE_in;
         wallMomentumY += f_BNW+f_TSE_in;
         wallMomentumZ -= f_BNW+f_TSE_in;
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_TNW_in=f_BSE - c1o216 * drho;
         wallMomentumX += f_BSE+f_TNW_in;
         wallMomentumY -= f_BSE+f_TNW_in;
         wallMomentumZ -= f_BSE+f_TNW_in;
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         f_BSE_in=f_TNW - c1o216 * drho;
         wallMomentumX -= f_TNW+f_BSE_in;
         wallMomentumY += f_TNW+f_BSE_in;
         wallMomentumZ += f_TNW+f_BSE_in;
      }

      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // //Compute wall velocity
      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      real VeloX=0.0, VeloY=0.0, VeloZ=0.0;

      q = q_dirB[k];
      real eps = 0.001f;

      iMEM( k, k_N[k],
         normalX, normalY, normalZ,
         vx, vy, vz,
         vx_el,      vy_el,      vz_el,
         vx_w_mean,  vy_w_mean,  vz_w_mean,
         vx1,        vx2,        vx3,
         c1o1+drho,
         samplingOffset,
         q,
         1.0,
         eps,
         z0,
         hasWallModelMonitor,
         u_star_monitor,
         wallMomentumX, wallMomentumY, wallMomentumZ,
         VeloX, VeloY, VeloZ);

      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      // //Add wall velocity and write f's
      // ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dM00])[kw] = f_W_in - (c6o1*c2o27*( VeloX     ));
         wallMomentumX += -(c6o1*c2o27*( VeloX     ));
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dP00])[ke] = f_E_in - (c6o1*c2o27*(-VeloX     ));
         wallMomentumX -= - (c6o1*c2o27*(-VeloX     ));
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[d0M0])[ks] = f_S_in - (c6o1*c2o27*( VeloY     ));
         wallMomentumY += - (c6o1*c2o27*( VeloY     ));
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[d0P0])[kn] = f_N_in - (c6o1*c2o27*(-VeloY     ));
         wallMomentumY -=  -(c6o1*c2o27*(-VeloY     ));
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[d00M])[kb] = f_B_in - (c6o1*c2o27*( VeloZ     ));
         wallMomentumZ += - (c6o1*c2o27*( VeloZ     ));
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[d00P])[kt] = f_T_in - (c6o1*c2o27*(-VeloZ     ));
         wallMomentumZ -= -(c6o1*c2o27*(-VeloZ     ));
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dMM0])[ksw] = f_SW_in - (c6o1*c1o54*(VeloX+VeloY));
         wallMomentumX +=  -(c6o1*c1o54*(VeloX+VeloY));
         wallMomentumY +=  -(c6o1*c1o54*(VeloX+VeloY));
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dPP0])[kne] = f_NE_in - (c6o1*c1o54*(-VeloX-VeloY));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX-VeloY));
         wallMomentumY -= - (c6o1*c1o54*(-VeloX-VeloY));
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dMP0])[knw] = f_NW_in - (c6o1*c1o54*( VeloX-VeloY));
         wallMomentumX += -(c6o1*c1o54*( VeloX-VeloY));
         wallMomentumY -= -(c6o1*c1o54*( VeloX-VeloY));
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dPM0])[kse] = f_SE_in - (c6o1*c1o54*(-VeloX+VeloY));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX+VeloY));
         wallMomentumY += - (c6o1*c1o54*(-VeloX+VeloY));
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dM0M])[kbw] = f_BW_in - (c6o1*c1o54*( VeloX+VeloZ));
         wallMomentumX += - (c6o1*c1o54*( VeloX+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( VeloX+VeloZ));
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dP0P])[kte] = f_TE_in - (c6o1*c1o54*(-VeloX-VeloZ));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*(-VeloX-VeloZ));
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dM0P])[ktw] = f_TW_in - (c6o1*c1o54*( VeloX-VeloZ));
         wallMomentumX += - (c6o1*c1o54*( VeloX-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( VeloX-VeloZ));
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dP0M])[kbe] = f_BE_in - (c6o1*c1o54*(-VeloX+VeloZ));
         wallMomentumX -= - (c6o1*c1o54*(-VeloX+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*(-VeloX+VeloZ));
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[d0MM])[kbs] = f_BS_in - (c6o1*c1o54*( VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o54*( VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( VeloY+VeloZ));
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[d0PP])[ktn] = f_TN_in - (c6o1*c1o54*( -VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o54*( -VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( -VeloY-VeloZ));
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[d0MP])[kts] = f_TS_in - (c6o1*c1o54*( VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o54*( VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o54*( VeloY-VeloZ));
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[d0PM])[kbn] = f_BN_in - (c6o1*c1o54*( -VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o54*( -VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o54*( -VeloY+VeloZ));
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dMMM])[kbsw] = f_BSW_in - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*( VeloX+VeloY+VeloZ));
      }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dPPP])[ktne] = f_TNE_in - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*(-VeloX-VeloY-VeloZ));
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dMMP])[ktsw] = f_TSW_in - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*( VeloX+VeloY-VeloZ));
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dPPM])[kbne] = f_BNE_in - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*(-VeloX-VeloY+VeloZ));
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dMPM])[kbnw] = f_BNW_in - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumY -= - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*( VeloX-VeloY+VeloZ));
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dPMP])[ktse] = f_TSE_in - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumY += - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*(-VeloX+VeloY-VeloZ));
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dMPP])[ktnw] = f_TNW_in - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumX += - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumY -= - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
         wallMomentumZ -= - (c6o1*c1o216*( VeloX-VeloY-VeloZ));
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         (D.f[dPMM])[kbse] = f_BSE_in - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumX -= - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumY += - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
         wallMomentumZ += - (c6o1*c1o216*(-VeloX+VeloY+VeloZ));
      }

      if(hasWallModelMonitor)
      {
         Fx_monitor[k] = wallMomentumX;
         Fy_monitor[k] = wallMomentumY;
         Fz_monitor[k] = wallMomentumZ;
      }

   }
}

//! \}
