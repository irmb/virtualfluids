#include "hip/hip_runtime.h"
//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
//  for more details.
//
//  SPDX-License-Identifier: GPL-3.0-or-later
//  SPDX-FileCopyrightText: Copyright © VirtualFluids Project contributors, see AUTHORS.md in root folder
//
//! \addtogroup gpu_BoundaryConditions BoundaryConditions
//! \ingroup gpu_core core
//! \{
//! \author Martin Schoenherr, Anna Wellmann
//======================================================================================
#include "Calculation/Calculation.h" 
#include "lbm/constants/D3Q27.h"
#include "basics/constants/NumericConstants.h"
#include "Utilities/KernelUtilities.h"

using namespace vf::basics::constant;
using namespace vf::lbm::dir;
using namespace vf::gpu;

__global__ void VelocityInterpolatedIncompressible_Device(
    real* vx,
    real* vy,
    real* vz,
    real* DD, 
    int* k_Q, 
    real* QQ,
    unsigned int numberOfBCnodes, 
    real om1, 
    unsigned int* neighborX,
    unsigned int* neighborY,
    unsigned int* neighborZ,
    unsigned long long numberOfLBnodes, 
    bool isEvenTimestep)
{
   Distributions27 D;
   if (isEvenTimestep==true)
   {
      D.f[dP00] = &DD[dP00 * numberOfLBnodes];
      D.f[dM00] = &DD[dM00 * numberOfLBnodes];
      D.f[d0P0] = &DD[d0P0 * numberOfLBnodes];
      D.f[d0M0] = &DD[d0M0 * numberOfLBnodes];
      D.f[d00P] = &DD[d00P * numberOfLBnodes];
      D.f[d00M] = &DD[d00M * numberOfLBnodes];
      D.f[dPP0] = &DD[dPP0 * numberOfLBnodes];
      D.f[dMM0] = &DD[dMM0 * numberOfLBnodes];
      D.f[dPM0] = &DD[dPM0 * numberOfLBnodes];
      D.f[dMP0] = &DD[dMP0 * numberOfLBnodes];
      D.f[dP0P] = &DD[dP0P * numberOfLBnodes];
      D.f[dM0M] = &DD[dM0M * numberOfLBnodes];
      D.f[dP0M] = &DD[dP0M * numberOfLBnodes];
      D.f[dM0P] = &DD[dM0P * numberOfLBnodes];
      D.f[d0PP] = &DD[d0PP * numberOfLBnodes];
      D.f[d0MM] = &DD[d0MM * numberOfLBnodes];
      D.f[d0PM] = &DD[d0PM * numberOfLBnodes];
      D.f[d0MP] = &DD[d0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[dPPP] = &DD[dPPP * numberOfLBnodes];
      D.f[dMMP] = &DD[dMMP * numberOfLBnodes];
      D.f[dPMP] = &DD[dPMP * numberOfLBnodes];
      D.f[dMPP] = &DD[dMPP * numberOfLBnodes];
      D.f[dPPM] = &DD[dPPM * numberOfLBnodes];
      D.f[dMMM] = &DD[dMMM * numberOfLBnodes];
      D.f[dPMM] = &DD[dPMM * numberOfLBnodes];
      D.f[dMPM] = &DD[dMPM * numberOfLBnodes];
   } 
   else
   {
      D.f[dM00] = &DD[dP00 * numberOfLBnodes];
      D.f[dP00] = &DD[dM00 * numberOfLBnodes];
      D.f[d0M0] = &DD[d0P0 * numberOfLBnodes];
      D.f[d0P0] = &DD[d0M0 * numberOfLBnodes];
      D.f[d00M] = &DD[d00P * numberOfLBnodes];
      D.f[d00P] = &DD[d00M * numberOfLBnodes];
      D.f[dMM0] = &DD[dPP0 * numberOfLBnodes];
      D.f[dPP0] = &DD[dMM0 * numberOfLBnodes];
      D.f[dMP0] = &DD[dPM0 * numberOfLBnodes];
      D.f[dPM0] = &DD[dMP0 * numberOfLBnodes];
      D.f[dM0M] = &DD[dP0P * numberOfLBnodes];
      D.f[dP0P] = &DD[dM0M * numberOfLBnodes];
      D.f[dM0P] = &DD[dP0M * numberOfLBnodes];
      D.f[dP0M] = &DD[dM0P * numberOfLBnodes];
      D.f[d0MM] = &DD[d0PP * numberOfLBnodes];
      D.f[d0PP] = &DD[d0MM * numberOfLBnodes];
      D.f[d0MP] = &DD[d0PM * numberOfLBnodes];
      D.f[d0PM] = &DD[d0MP * numberOfLBnodes];
      D.f[d000] = &DD[d000 * numberOfLBnodes];
      D.f[dPPP] = &DD[dMMM * numberOfLBnodes];
      D.f[dMMP] = &DD[dPPM * numberOfLBnodes];
      D.f[dPMP] = &DD[dMPM * numberOfLBnodes];
      D.f[dMPP] = &DD[dPMM * numberOfLBnodes];
      D.f[dPPM] = &DD[dMMP * numberOfLBnodes];
      D.f[dMMM] = &DD[dPPP * numberOfLBnodes];
      D.f[dPMM] = &DD[dMPP * numberOfLBnodes];
      D.f[dMPM] = &DD[dPMP * numberOfLBnodes];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfBCnodes)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real VeloX = vx[k];
      real VeloY = vy[k];
      real VeloZ = vz[k]; //(16.0*(u0*2.0)*bbx*bby*(grid_nx-bbx)*(grid_ny-bby))/(grid_nx*grid_nx*grid_ny*grid_ny)
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dP00 * numberOfBCnodes];
      q_dirW   = &QQ[dM00 * numberOfBCnodes];
      q_dirN   = &QQ[d0P0 * numberOfBCnodes];
      q_dirS   = &QQ[d0M0 * numberOfBCnodes];
      q_dirT   = &QQ[d00P * numberOfBCnodes];
      q_dirB   = &QQ[d00M * numberOfBCnodes];
      q_dirNE  = &QQ[dPP0 * numberOfBCnodes];
      q_dirSW  = &QQ[dMM0 * numberOfBCnodes];
      q_dirSE  = &QQ[dPM0 * numberOfBCnodes];
      q_dirNW  = &QQ[dMP0 * numberOfBCnodes];
      q_dirTE  = &QQ[dP0P * numberOfBCnodes];
      q_dirBW  = &QQ[dM0M * numberOfBCnodes];
      q_dirBE  = &QQ[dP0M * numberOfBCnodes];
      q_dirTW  = &QQ[dM0P * numberOfBCnodes];
      q_dirTN  = &QQ[d0PP * numberOfBCnodes];
      q_dirBS  = &QQ[d0MM * numberOfBCnodes];
      q_dirBN  = &QQ[d0PM * numberOfBCnodes];
      q_dirTS  = &QQ[d0MP * numberOfBCnodes];
      q_dirTNE = &QQ[dPPP * numberOfBCnodes];
      q_dirTSW = &QQ[dMMP * numberOfBCnodes];
      q_dirTSE = &QQ[dPMP * numberOfBCnodes];
      q_dirTNW = &QQ[dMPP * numberOfBCnodes];
      q_dirBNE = &QQ[dPPM * numberOfBCnodes];
      q_dirBSW = &QQ[dMMM * numberOfBCnodes];
      q_dirBSE = &QQ[dPMM * numberOfBCnodes];
      q_dirBNW = &QQ[dMPM * numberOfBCnodes];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dP00])[ke   ];
      f_E    = (D.f[dM00])[kw   ];
      f_S    = (D.f[d0P0])[kn   ];
      f_N    = (D.f[d0M0])[ks   ];
      f_B    = (D.f[d00P])[kt   ];
      f_T    = (D.f[d00M])[kb   ];
      f_SW   = (D.f[dPP0])[kne  ];
      f_NE   = (D.f[dMM0])[ksw  ];
      f_NW   = (D.f[dPM0])[kse  ];
      f_SE   = (D.f[dMP0])[knw  ];
      f_BW   = (D.f[dP0P])[kte  ];
      f_TE   = (D.f[dM0M])[kbw  ];
      f_TW   = (D.f[dP0M])[kbe  ];
      f_BE   = (D.f[dM0P])[ktw  ];
      f_BS   = (D.f[d0PP])[ktn  ];
      f_TN   = (D.f[d0MM])[kbs  ];
      f_TS   = (D.f[d0PM])[kbn  ];
      f_BN   = (D.f[d0MP])[kts  ];
      f_BSW  = (D.f[dPPP])[ktne ];
      f_BNE  = (D.f[dMMP])[ktsw ];
      f_BNW  = (D.f[dPMP])[ktse ];
      f_BSE  = (D.f[dMPP])[ktnw ];
      f_TSW  = (D.f[dPPM])[kbne ];
      f_TNE  = (D.f[dMMM])[kbsw ];
      f_TNW  = (D.f[dPMM])[kbse ];
      f_TSE  = (D.f[dMPM])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[d000])[kzero]); 

      vx1    =  ((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W); 
         

      vx2    =   (-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S); 

      vx3    =   ((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      //////////////////////////////////////////////////////////////////////////
      if (isEvenTimestep==false)
      {
         D.f[dP00] = &DD[dP00 * numberOfLBnodes];
         D.f[dM00] = &DD[dM00 * numberOfLBnodes];
         D.f[d0P0] = &DD[d0P0 * numberOfLBnodes];
         D.f[d0M0] = &DD[d0M0 * numberOfLBnodes];
         D.f[d00P] = &DD[d00P * numberOfLBnodes];
         D.f[d00M] = &DD[d00M * numberOfLBnodes];
         D.f[dPP0] = &DD[dPP0 * numberOfLBnodes];
         D.f[dMM0] = &DD[dMM0 * numberOfLBnodes];
         D.f[dPM0] = &DD[dPM0 * numberOfLBnodes];
         D.f[dMP0] = &DD[dMP0 * numberOfLBnodes];
         D.f[dP0P] = &DD[dP0P * numberOfLBnodes];
         D.f[dM0M] = &DD[dM0M * numberOfLBnodes];
         D.f[dP0M] = &DD[dP0M * numberOfLBnodes];
         D.f[dM0P] = &DD[dM0P * numberOfLBnodes];
         D.f[d0PP] = &DD[d0PP * numberOfLBnodes];
         D.f[d0MM] = &DD[d0MM * numberOfLBnodes];
         D.f[d0PM] = &DD[d0PM * numberOfLBnodes];
         D.f[d0MP] = &DD[d0MP * numberOfLBnodes];
         D.f[d000] = &DD[d000 * numberOfLBnodes];
         D.f[dPPP] = &DD[dPPP * numberOfLBnodes];
         D.f[dMMP] = &DD[dMMP * numberOfLBnodes];
         D.f[dPMP] = &DD[dPMP * numberOfLBnodes];
         D.f[dMPP] = &DD[dMPP * numberOfLBnodes];
         D.f[dPPM] = &DD[dPPM * numberOfLBnodes];
         D.f[dMMM] = &DD[dMMM * numberOfLBnodes];
         D.f[dPMM] = &DD[dPMM * numberOfLBnodes];
         D.f[dMPM] = &DD[dMPM * numberOfLBnodes];
      } 
      else
      {
         D.f[dM00] = &DD[dP00 * numberOfLBnodes];
         D.f[dP00] = &DD[dM00 * numberOfLBnodes];
         D.f[d0M0] = &DD[d0P0 * numberOfLBnodes];
         D.f[d0P0] = &DD[d0M0 * numberOfLBnodes];
         D.f[d00M] = &DD[d00P * numberOfLBnodes];
         D.f[d00P] = &DD[d00M * numberOfLBnodes];
         D.f[dMM0] = &DD[dPP0 * numberOfLBnodes];
         D.f[dPP0] = &DD[dMM0 * numberOfLBnodes];
         D.f[dMP0] = &DD[dPM0 * numberOfLBnodes];
         D.f[dPM0] = &DD[dMP0 * numberOfLBnodes];
         D.f[dM0M] = &DD[dP0P * numberOfLBnodes];
         D.f[dP0P] = &DD[dM0M * numberOfLBnodes];
         D.f[dM0P] = &DD[dP0M * numberOfLBnodes];
         D.f[dP0M] = &DD[dM0P * numberOfLBnodes];
         D.f[d0MM] = &DD[d0PP * numberOfLBnodes];
         D.f[d0PP] = &DD[d0MM * numberOfLBnodes];
         D.f[d0MP] = &DD[d0PM * numberOfLBnodes];
         D.f[d0PM] = &DD[d0MP * numberOfLBnodes];
         D.f[d000] = &DD[d000 * numberOfLBnodes];
         D.f[dPPP] = &DD[dMMM * numberOfLBnodes];
         D.f[dMMP] = &DD[dPPM * numberOfLBnodes];
         D.f[dPMP] = &DD[dMPM * numberOfLBnodes];
         D.f[dMPP] = &DD[dPMM * numberOfLBnodes];
         D.f[dPPM] = &DD[dMMP * numberOfLBnodes];
         D.f[dMMM] = &DD[dPPP * numberOfLBnodes];
         D.f[dPMM] = &DD[dMPP * numberOfLBnodes];
         D.f[dMPM] = &DD[dPMP * numberOfLBnodes];
      }

      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

      q = q_dirE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho+c9o2*( vx1        )*( vx1        )-cu_sq); 
         (D.f[dM00])[kw]=(c1o1-q)/(c1o1+q)*(f_E-f_W+(f_E+f_W-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_E+f_W)-c6o1*c2o27*( VeloX     ))/(c1o1+q);
      }

      q = q_dirW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho+c9o2*(-vx1        )*(-vx1        )-cu_sq); 
         (D.f[dP00])[ke]=(c1o1-q)/(c1o1+q)*(f_W-f_E+(f_W+f_E-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_W+f_E)-c6o1*c2o27*(-VeloX     ))/(c1o1+q);
      }

      q = q_dirN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho+c9o2*(     vx2    )*(     vx2    )-cu_sq); 
         (D.f[d0M0])[ks]=(c1o1-q)/(c1o1+q)*(f_N-f_S+(f_N+f_S-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_N+f_S)-c6o1*c2o27*( VeloY     ))/(c1o1+q);
      }

      q = q_dirS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho+c9o2*(    -vx2    )*(    -vx2    )-cu_sq); 
         (D.f[d0P0])[kn]=(c1o1-q)/(c1o1+q)*(f_S-f_N+(f_S+f_N-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_S+f_N)-c6o1*c2o27*(-VeloY     ))/(c1o1+q);
      }

      q = q_dirT[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho+c9o2*(         vx3)*(         vx3)-cu_sq); 
         (D.f[d00M])[kb]=(c1o1-q)/(c1o1+q)*(f_T-f_B+(f_T+f_B-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_T+f_B)-c6o1*c2o27*( VeloZ     ))/(c1o1+q);
      }

      q = q_dirB[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c2o27* (drho+c9o2*(        -vx3)*(        -vx3)-cu_sq); 
         (D.f[d00P])[kt]=(c1o1-q)/(c1o1+q)*(f_B-f_T+(f_B+f_T-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_B+f_T)-c6o1*c2o27*(-VeloZ     ))/(c1o1+q);
      }

      q = q_dirNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         (D.f[dMM0])[ksw]=(c1o1-q)/(c1o1+q)*(f_NE-f_SW+(f_NE+f_SW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NE+f_SW)-c6o1*c1o54*(VeloX+VeloY))/(c1o1+q);
      }

      q = q_dirSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         (D.f[dPP0])[kne]=(c1o1-q)/(c1o1+q)*(f_SW-f_NE+(f_SW+f_NE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SW+f_NE)-c6o1*c1o54*(-VeloX-VeloY))/(c1o1+q);
      }

      q = q_dirSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         (D.f[dMP0])[knw]=(c1o1-q)/(c1o1+q)*(f_SE-f_NW+(f_SE+f_NW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_SE+f_NW)-c6o1*c1o54*( VeloX-VeloY))/(c1o1+q);
      }

      q = q_dirNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         (D.f[dPM0])[kse]=(c1o1-q)/(c1o1+q)*(f_NW-f_SE+(f_NW+f_SE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_NW+f_SE)-c6o1*c1o54*(-VeloX+VeloY))/(c1o1+q);
      }

      q = q_dirTE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         (D.f[dM0M])[kbw]=(c1o1-q)/(c1o1+q)*(f_TE-f_BW+(f_TE+f_BW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TE+f_BW)-c6o1*c1o54*( VeloX+VeloZ))/(c1o1+q);
      }

      q = q_dirBW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         (D.f[dP0P])[kte]=(c1o1-q)/(c1o1+q)*(f_BW-f_TE+(f_BW+f_TE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BW+f_TE)-c6o1*c1o54*(-VeloX-VeloZ))/(c1o1+q);
      }

      q = q_dirBE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         (D.f[dM0P])[ktw]=(c1o1-q)/(c1o1+q)*(f_BE-f_TW+(f_BE+f_TW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BE+f_TW)-c6o1*c1o54*( VeloX-VeloZ))/(c1o1+q);
      }

      q = q_dirTW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         (D.f[dP0M])[kbe]=(c1o1-q)/(c1o1+q)*(f_TW-f_BE+(f_TW+f_BE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TW+f_BE)-c6o1*c1o54*(-VeloX+VeloZ))/(c1o1+q);
      }

      q = q_dirTN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         (D.f[d0MM])[kbs]=(c1o1-q)/(c1o1+q)*(f_TN-f_BS+(f_TN+f_BS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TN+f_BS)-c6o1*c1o54*( VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirBS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         (D.f[d0PP])[ktn]=(c1o1-q)/(c1o1+q)*(f_BS-f_TN+(f_BS+f_TN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BS+f_TN)-c6o1*c1o54*( -VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBN[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         (D.f[d0MP])[kts]=(c1o1-q)/(c1o1+q)*(f_BN-f_TS+(f_BN+f_TS-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BN+f_TS)-c6o1*c1o54*( VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTS[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o54* (drho+c9o2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         (D.f[d0PM])[kbn]=(c1o1-q)/(c1o1+q)*(f_TS-f_BN+(f_TS+f_BN-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TS+f_BN)-c6o1*c1o54*( -VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirTNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho+c9o2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         (D.f[dMMM])[kbsw]=(c1o1-q)/(c1o1+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNE+f_BSW)-c6o1*c1o216*( VeloX+VeloY+VeloZ))/(c1o1+q);
     }

      q = q_dirBSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho+c9o2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         (D.f[dPPP])[ktne]=(c1o1-q)/(c1o1+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSW+f_TNE)-c6o1*c1o216*(-VeloX-VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBNE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho+c9o2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         (D.f[dMMP])[ktsw]=(c1o1-q)/(c1o1+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNE+f_TSW)-c6o1*c1o216*( VeloX+VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTSW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho+c9o2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         (D.f[dPPM])[kbne]=(c1o1-q)/(c1o1+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSW+f_BNE)-c6o1*c1o216*(-VeloX-VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirTSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho+c9o2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         (D.f[dMPM])[kbnw]=(c1o1-q)/(c1o1+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TSE+f_BNW)-c6o1*c1o216*( VeloX-VeloY+VeloZ))/(c1o1+q);
      }

      q = q_dirBNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho+c9o2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         (D.f[dPMP])[ktse]=(c1o1-q)/(c1o1+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BNW+f_TSE)-c6o1*c1o216*(-VeloX+VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirBSE[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho+c9o2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         (D.f[dMPP])[ktnw]=(c1o1-q)/(c1o1+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_BSE+f_TNW)-c6o1*c1o216*( VeloX-VeloY-VeloZ))/(c1o1+q);
      }

      q = q_dirTNW[k];
      if (q>=c0o1 && q<=c1o1)
      {
         feq=c1o216*(drho+c9o2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         (D.f[dPMM])[kbse]=(c1o1-q)/(c1o1+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-c2o1*feq*om1)/(c1o1-om1))*c1o2+(q*(f_TNW+f_BSE)-c6o1*c1o216*(-VeloX+VeloY+VeloZ))/(c1o1+q);
      }
   }
}

//! \}
