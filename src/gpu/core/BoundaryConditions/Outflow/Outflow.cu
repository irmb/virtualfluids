//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ /
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "LBM/LB.h"
#include <cuda_helper/CudaGrid.h>

#include "BoundaryConditions/Outflow/Outflow_Device.cuh"
#include "Parameter/Parameter.h"

void OutflowNonReflecting(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
    dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
    dim3 threads(parameterDevice->numberofthreads, 1, 1 );

    OutflowNonReflecting_Device<<< grid, threads >>> (
        boundaryCondition->RhoBC,
        parameterDevice->distributions.f[0],
        boundaryCondition->k,
        boundaryCondition->kN,
        boundaryCondition->numberOfBCnodes,
        parameterDevice->omega,
        parameterDevice->neighborX,
        parameterDevice->neighborY,
        parameterDevice->neighborZ,
        parameterDevice->numberOfNodes,
        parameterDevice->isEvenTimestep,
        vf::lbm::dir::dP00);
    getLastCudaError("OutflowNonReflecting_Device execution failed");
}

void OutflowNonReflectingPressureCorrection(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
    dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
    dim3 threads(parameterDevice->numberofthreads, 1, 1 );

    OutflowNonReflectingPressureCorrection_Device<<< grid, threads >>> (
        boundaryCondition->RhoBC,
        parameterDevice->distributions.f[0],
        boundaryCondition->k,
        boundaryCondition->kN,
        boundaryCondition->numberOfBCnodes,
        parameterDevice->omega,
        parameterDevice->neighborX,
        parameterDevice->neighborY,
        parameterDevice->neighborZ,
        parameterDevice->numberOfNodes,
        parameterDevice->isEvenTimestep,
        vf::lbm::dir::dP00,
        parameterDevice->outflowPressureCorrectionFactor);
    getLastCudaError("OutflowNonReflectingPressureCorrection_Device execution failed");
}
