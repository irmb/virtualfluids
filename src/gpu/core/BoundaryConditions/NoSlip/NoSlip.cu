//=======================================================================================
// ____          ____    __    ______     __________   __      __       __        __
// \    \       |    |  |  |  |   _   \  |___    ___| |  |    |  |     /  \      |  |
//  \    \      |    |  |  |  |  |_)   |     |  |     |  |    |  |    /    \     |  |
//   \    \     |    |  |  |  |   _   /      |  |     |  |    |  |   /  /\  \    |  |
//    \    \    |    |  |  |  |  | \  \      |  |     |   \__/   |  /  ____  \   |  |____
//     \    \   |    |  |__|  |__|  \__\     |__|      \________/  /__/    \__\  |_______|
//      \    \  |    |   ________________________________________________________________
//       \    \ |    |  |  ______________________________________________________________|
//        \    \|    |  |  |         __          __     __     __     ______      _______
//         \         |  |  |_____   |  |        |  |   |  |   |  |   |   _  \    /  _____)
//          \        |  |   _____|  |  |        |  |   |  |   |  |   |  | \  \   \_______
//           \       |  |  |        |  |_____   |   \_/   |   |  |   |  |_/  /    _____  |
//            \ _____|  |__|        |________|   \_______/    |__|   |______/    (_______/
//
//  This file is part of VirtualFluids. VirtualFluids is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  VirtualFluids is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with VirtualFluids (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \author Martin Schoenherr
//=======================================================================================
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "LBM/LB.h"
#include <cuda_helper/CudaGrid.h>

#include "BoundaryConditions/NoSlip/NoSlip_Device.cuh"
#include "Parameter/Parameter.h"

void NoSlipBounceBack(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
    dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
    dim3 threads(parameterDevice->numberofthreads, 1, 1 );

    NoSlipBounceBack_Device<<< grid, threads >>> (
        parameterDevice->distributions.f[0],
        boundaryCondition->k,
        boundaryCondition->q27[0],
        boundaryCondition->numberOfBCnodes,
        parameterDevice->neighborX,
        parameterDevice->neighborY,
        parameterDevice->neighborZ,
        parameterDevice->numberOfNodes,
        parameterDevice->isEvenTimestep);
    getLastCudaError("NoSlipBounceBack_Device execution failed");
}

void NoSlipInterpolatedIncompressible(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
    dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
    dim3 threads(parameterDevice->numberofthreads, 1, 1 );

    NoSlipInterpolatedIncompressible_Device<<< grid, threads >>> (
        parameterDevice->distributions.f[0],
        boundaryCondition->k,
        boundaryCondition->q27[0],
        boundaryCondition->numberOfBCnodes,
        parameterDevice->omega,
        parameterDevice->neighborX,
        parameterDevice->neighborY,
        parameterDevice->neighborZ,
        parameterDevice->numberOfNodes,
        parameterDevice->isEvenTimestep);
      getLastCudaError("NoSlipInterpolatedIncompressible_Device execution failed");
}

void NoSlipInterpolatedCompressible(LBMSimulationParameter* parameterDevice, QforBoundaryConditions* boundaryCondition)
{
    dim3 grid = vf::cuda::getCudaGrid( parameterDevice->numberofthreads,  boundaryCondition->numberOfBCnodes);
    dim3 threads(parameterDevice->numberofthreads, 1, 1 );

    NoSlipInterpolatedCompressible_Device<<< grid, threads >>> (
        parameterDevice->distributions.f[0],
        boundaryCondition->k,
        boundaryCondition->q27[0],
        boundaryCondition->numberOfBCnodes,
        parameterDevice->omega,
        parameterDevice->neighborX,
        parameterDevice->neighborY,
        parameterDevice->neighborZ,
        parameterDevice->numberOfNodes,
        parameterDevice->isEvenTimestep);
    getLastCudaError("NoSlipInterpolatedCompressible_Device execution failed");
}

