#include "hip/hip_runtime.h"
#include "Sphere.h"

#include <algorithm>    // std::min

#include "geometries/Vertex/Vertex.h"

Sphere::Sphere(const double& centerX, const double& centerY, const double& centerZ, const double& radius)
    : centerX(centerX), centerY(centerY), centerZ(centerZ), radius(radius)
{

}

Sphere::~Sphere()
{
}

SPtr<Sphere> Sphere::makeShared(double centerX, double centerY, double centerZ, double radius)
{
    return SPtr<Sphere>(new Sphere(centerX, centerY, centerZ, radius));
}

Object* Sphere::clone() const
{
    return new Sphere(centerX, centerY, centerZ, radius);
}

double Sphere::getX1Centroid()
{
    return centerX;
}

double Sphere::getX1Minimum()
{
    return centerX - radius;
}

double Sphere::getX1Maximum()
{
    return centerX + radius;
}

double Sphere::getX2Centroid()
{
    return centerY;
}

double Sphere::getX2Minimum()
{
    return centerY - radius;
}

double Sphere::getX2Maximum()
{
    return centerY + radius;
}

double Sphere::getX3Centroid()
{
    return centerZ;
}

double Sphere::getX3Minimum()
{
    return centerZ - radius;
}

double Sphere::getX3Maximum()
{
    return centerZ + radius;
}

bool Sphere::isPointInObject(const double& x1, const double& x2, const double& x3, const double& minOffset,
    const double& maxOffset)
{
    double offset = maxOffset;
    if (x1 < centerX || x2 < centerY || x3 < centerZ)
        offset = minOffset;
        

    const double deltaX1 = x1 - centerX;
    const double deltaX2 = x2 - centerY;
    const double deltaX3 = x3 - centerZ;

    return (deltaX1*deltaX1 + deltaX2*deltaX2 + deltaX3*deltaX3) < ((this->radius - offset) * (this->radius - offset));
}


void Sphere::scale(double delta)
{
    this->radius += delta;
}

CUDA_HOST int Sphere::getIntersection(const Vertex & point, const Vertex & direction, Vertex & pointOnObject, real & qVal)
{
    
    Vertex relativePoint( point.x - this->centerX, 
                          point.y - this->centerY, 
                          point.z - this->centerZ );

    real directionSquare = direction.x * direction.x
                         + direction.y * direction.y
                         + direction.z * direction.z;

    real p = 2* ( relativePoint.x * direction.x
                + relativePoint.y * direction.y
                + relativePoint.z * direction.z )
                / directionSquare;

    real q = ( relativePoint.x * relativePoint.x
             + relativePoint.y * relativePoint.y
             + relativePoint.z * relativePoint.z
             - (real)this->radius * (real)this->radius )
           / directionSquare;

    real discriminant = 0.25 * p * p - q;


    if( discriminant < 0.0 ) return 1;

    real result1 = - 0.5 * p + std::sqrt( discriminant );
    real result2 = - 0.5 * p - std::sqrt( discriminant );

    if( result1 < 0.0 && result2 < 0.0 ) return 1;

    if (result1 < 0.0)
        result1 = (real)FLT_MAX;
    if (result2 < 0.0)
        result2 = (real)FLT_MAX;

    real t = std::min( result1, result2 );

    pointOnObject.x = point.x + t * direction.x;
    pointOnObject.y = point.y + t * direction.y;
    pointOnObject.z = point.z + t * direction.z;

    qVal = t;

    return 0;
}
