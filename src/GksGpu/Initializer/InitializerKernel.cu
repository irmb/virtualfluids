#include "hip/hip_runtime.h"
#include "Initializer.h"

#include <sstream>
#define _USE_MATH_DEFINES
#include <math.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Core/PointerDefinitions.h"
#include "Core/RealConstants.h"

#include "DataBase/DataBaseStruct.h"

#include "Definitions/MemoryAccessPattern.h"
#include "Definitions/PassiveScalar.h"

#include "CudaUtility/CudaRunKernel.hpp"

__global__                 void initializeDataUpdateKernel  ( DataBaseStruct dataBase, uint numberOfEntities );

__host__ __device__ inline void initializeDataUpdateFunction( DataBaseStruct dataBase, uint index );

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Initializer::initializeDataUpdate( std::shared_ptr<DataBase> dataBase )
{
    CudaUtility::CudaGrid grid( dataBase->numberOfCells, 32 );

    runKernel( initializeDataUpdateKernel,
               initializeDataUpdateFunction,
               dataBase->getDeviceType(), grid, 
               dataBase->toStruct() );
}

__global__ void initializeDataUpdateKernel(DataBaseStruct dataBase, uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index > numberOfEntities ) return;

    initializeDataUpdateFunction( dataBase, index );
}

__host__ __device__ inline void initializeDataUpdateFunction(DataBaseStruct dataBase, uint index)
{
    dataBase.dataUpdate[ RHO__(index, dataBase.numberOfCells) ] = zero;
    dataBase.dataUpdate[ RHO_U(index, dataBase.numberOfCells) ] = zero;
    dataBase.dataUpdate[ RHO_V(index, dataBase.numberOfCells) ] = zero;
    dataBase.dataUpdate[ RHO_E(index, dataBase.numberOfCells) ] = zero;
#ifdef USE_PASSIVE_SCALAR
	dataBase.dataUpdate[ RHO_S(index, dataBase.numberOfCells) ] = zero;
#endif // USE_PASSIVE_SCALAR

    dataBase.massFlux[ VEC_X(index, dataBase.numberOfCells) ]   = zero;
    dataBase.massFlux[ VEC_Y(index, dataBase.numberOfCells) ]   = zero;
}
