#include "hip/hip_runtime.h"
#include "Initializer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Core/PointerDefinitions.h"
#include "Core/RealConstants.h"

#include "DataBase/DataBaseStruct.h"

#include "Definitions/MemoryAccessPattern.h"
#include "Definitions/PassiveScalar.h"

#include "CudaUtility/CudaRunKernel.hpp"

__global__                 void initializeDataUpdateKernel  ( DataBaseStruct dataBase, uint numberOfEntities );

__host__ __device__ inline void initializeDataUpdateFunction( DataBaseStruct dataBase, uint index );

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Initializer::initializeDataUpdate( SPtr<DataBase> dataBase )
{
    CudaUtility::CudaGrid grid( dataBase->numberOfCells, 32 );

    runKernel( initializeDataUpdateKernel,
               initializeDataUpdateFunction,
               dataBase->getDeviceType(), grid, 
               dataBase->toStruct() );

    hipDeviceSynchronize();

    getLastCudaError("Initializer::initializeDataUpdate( SPtr<DataBase> dataBase )");
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void initializeDataUpdateKernel(DataBaseStruct dataBase, uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index >= numberOfEntities ) return;

    initializeDataUpdateFunction( dataBase, index );
}

__host__ __device__ inline void initializeDataUpdateFunction(DataBaseStruct dataBase, uint index)
{
    dataBase.dataUpdate[ RHO__(index, dataBase.numberOfCells) ] = c0o1;
    dataBase.dataUpdate[ RHO_U(index, dataBase.numberOfCells) ] = c0o1;
    dataBase.dataUpdate[ RHO_V(index, dataBase.numberOfCells) ] = c0o1;
    dataBase.dataUpdate[ RHO_W(index, dataBase.numberOfCells) ] = c0o1;
    dataBase.dataUpdate[ RHO_E(index, dataBase.numberOfCells) ] = c0o1;
#ifdef USE_PASSIVE_SCALAR
	dataBase.dataUpdate[ RHO_S_1(index, dataBase.numberOfCells) ] = c0o1;
	dataBase.dataUpdate[ RHO_S_2(index, dataBase.numberOfCells) ] = c0o1;
#endif // USE_PASSIVE_SCALAR

    dataBase.massFlux[ VEC_X(index, dataBase.numberOfCells) ]   = c0o1;
    dataBase.massFlux[ VEC_Y(index, dataBase.numberOfCells) ]   = c0o1;
    dataBase.massFlux[ VEC_Z(index, dataBase.numberOfCells) ]   = c0o1;

    dataBase.diffusivity[ index ] = c1o1;
}
