#include "hip/hip_runtime.h"
#include "Inflow.h"

#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Core/PointerDefinitions.h"
#include "Core/RealConstants.h"

#include "DataBase/DataBase.h"
#include "DataBase/DataBaseStruct.h"

#include "Definitions/MemoryAccessPattern.h"
#include "Definitions/PassiveScalar.h"

#include "FlowStateData/FlowStateData.cuh"
#include "FlowStateData/AccessDeviceData.cuh"

#include "CudaUtility/CudaRunKernel.hpp"

//////////////////////////////////////////////////////////////////////////

__global__                 void boundaryConditionKernel  ( const DataBaseStruct dataBase, 
                                                           const InflowStruct boundaryCondition, 
                                                           const Parameters parameters,
                                                           const uint startIndex,
                                                           const uint numberOfEntities );

__host__ __device__ inline void boundaryConditionFunction( const DataBaseStruct& dataBase, 
                                                           const InflowStruct& boundaryCondition, 
                                                           const Parameters& parameters,
                                                           const uint startIndex,
                                                           const uint index );

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

void Inflow::runBoundaryConditionKernel(const SPtr<DataBase> dataBase, 
                                                const Parameters parameters, 
                                                const uint level)
{    
    CudaUtility::CudaGrid grid( this->numberOfCellsPerLevel[ level ], 32 );

    runKernel( boundaryConditionKernel,
               boundaryConditionFunction,
               dataBase->getDeviceType(), grid, 
               dataBase->toStruct(),
               this->toStruct(),
               parameters,
               this->startOfCellsPerLevel[ level ] );

    getLastCudaError("IsothermalWall::runBoundaryConditionKernel( const SPtr<DataBase> dataBase, const Parameters parameters, const uint level )");
}

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

__global__ void boundaryConditionKernel(const DataBaseStruct dataBase, 
                                        const InflowStruct boundaryCondition, 
                                        const Parameters parameters,
                                        const uint startIndex,
                                        const uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index >= numberOfEntities ) return;

    boundaryConditionFunction( dataBase, boundaryCondition, parameters, startIndex, index );
}

__host__ __device__ inline void boundaryConditionFunction(const DataBaseStruct& dataBase, 
                                                          const InflowStruct& boundaryCondition, 
                                                          const Parameters& parameters,
                                                          const uint startIndex,
                                                          const uint index)
{
    uint ghostCellIdx  = boundaryCondition.ghostCells [ startIndex + index ];
    uint domainCellIdx = boundaryCondition.domainCells[ startIndex + index ];
    uint secondCellIdx = boundaryCondition.secondCells[ startIndex + index ];

    PrimitiveVariables ghostCellPrim;
    {
        PrimitiveVariables domainCellPrim;
        PrimitiveVariables secondCellPrim;

        {
            ConservedVariables domainCellData;
            readCellData( domainCellIdx, dataBase, domainCellData );
            domainCellPrim = toPrimitiveVariables( domainCellData, parameters.K );
        }

        real factor;
        {
            real y = dataBase.cellCenter[ VEC_Y(ghostCellIdx, dataBase.numberOfCells) ];

            factor =  ( boundaryCondition.a0 
                      + boundaryCondition.a1*y 
                      + boundaryCondition.a2*y*y  ) * ( four / boundaryCondition.a1 / boundaryCondition.a1 );

            factor = one;
        }

        //ghostCellPrim.rho    = two *          boundaryCondition.rho        - domainCellPrim.rho;
        ghostCellPrim.U      = two * factor * boundaryCondition.velocity.x - domainCellPrim.U;
        ghostCellPrim.V      = two * factor * boundaryCondition.velocity.y - domainCellPrim.V;
        ghostCellPrim.W      = two * factor * boundaryCondition.velocity.z - domainCellPrim.W;
        ghostCellPrim.lambda = two *          boundaryCondition.lambda     - domainCellPrim.lambda;
    #ifdef USE_PASSIVE_SCALAR
        ghostCellPrim.S_1    = two *          boundaryCondition.S_1        - domainCellPrim.S_1;
        ghostCellPrim.S_2    = two *          boundaryCondition.S_2        - domainCellPrim.S_2;
    #endif // USE_PASSIVE_SCALAR
        
        real p = c1o2 * domainCellPrim.rho / domainCellPrim.lambda;
        ghostCellPrim.rho = two * p * ghostCellPrim.lambda;
    }

    {
        ConservedVariables ghostCons = toConservedVariables( ghostCellPrim, parameters.K );

        writeCellData( ghostCellIdx, dataBase, ghostCons );
    }
}

Inflow::Inflow(SPtr<DataBase> dataBase, Vec3 velocity, real lambda, real rho, real S, real a0, real a1, real a2)
    : BoundaryCondition( dataBase )
{
    this->velocity       = velocity;
    this->lambda         = lambda;
    this->rho            = rho;
    this->S_1            = S_1;
    this->S_2            = S_2;

    this->a0             = a0;
    this->a1             = a1;
    this->a2             = a2;
}

bool Inflow::isWall()
{
    return false;
}

bool Inflow::secondCellsNeeded()
{
    return false;
}

