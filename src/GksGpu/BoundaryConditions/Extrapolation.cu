#include "hip/hip_runtime.h"
#include "Extrapolation.h"

#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Core/PointerDefinitions.h"
#include "Core/RealConstants.h"

#include "DataBase/DataBase.h"
#include "DataBase/DataBaseStruct.h"

#include "Definitions/MemoryAccessPattern.h"
#include "Definitions/PassiveScalar.h"

#include "FlowStateData/FlowStateData.cuh"
#include "FlowStateData/AccessDeviceData.cuh"

#include "CudaUtility/CudaRunKernel.hpp"

namespace GksGpu{

//////////////////////////////////////////////////////////////////////////

__global__                 void boundaryConditionKernel  ( const DataBaseStruct dataBase, 
                                                           const ExtrapolationStruct boundaryCondition, 
                                                           const Parameters parameters,
                                                           const uint startIndex,
                                                           const uint numberOfEntities );

__host__ __device__ inline void boundaryConditionFunction( const DataBaseStruct& dataBase, 
                                                           const ExtrapolationStruct& boundaryCondition, 
                                                           const Parameters& parameters,
                                                           const uint startIndex,
                                                           const uint index );

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

void Extrapolation::runBoundaryConditionKernel(const SPtr<DataBase> dataBase, 
                                               const Parameters parameters, 
                                               const uint level)
{    
    CudaUtility::CudaGrid grid( this->numberOfCellsPerLevel[ level ], 32 );

    runKernel( boundaryConditionKernel,
               boundaryConditionFunction,
               dataBase->getDeviceType(), grid, 
               dataBase->toStruct(),
               this->toStruct(),
               parameters,
               this->startOfCellsPerLevel[ level ] );

    getLastCudaError("IsothermalWall::runBoundaryConditionKernel( const SPtr<DataBase> dataBase, const Parameters parameters, const uint level )");
}

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

__global__ void boundaryConditionKernel(const DataBaseStruct dataBase, 
                                        const ExtrapolationStruct boundaryCondition, 
                                        const Parameters parameters,
                                        const uint startIndex,
                                        const uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index >= numberOfEntities ) return;

    boundaryConditionFunction( dataBase, boundaryCondition, parameters, startIndex, index );
}

__host__ __device__ inline void boundaryConditionFunction(const DataBaseStruct& dataBase, 
                                                          const ExtrapolationStruct& boundaryCondition, 
                                                          const Parameters& parameters,
                                                          const uint startIndex,
                                                          const uint index)
{
    uint ghostCellIdx  = boundaryCondition.ghostCells [ startIndex + index ];
    uint domainCellIdx = boundaryCondition.domainCells[ startIndex + index ];

    ConservedVariables domainCellData;
    readCellData ( domainCellIdx, dataBase, domainCellData );
    writeCellData( ghostCellIdx , dataBase, domainCellData );
}

Extrapolation::Extrapolation(SPtr<DataBase> dataBase)
    : BoundaryCondition( dataBase )
{
}

bool Extrapolation::isWall()
{
    return false;
}

bool Extrapolation::secondCellsNeeded()
{
    return false;
}

} // namespace GksGpu

