#include "hip/hip_runtime.h"
#include "Pressure.h"

#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Core/PointerDefinitions.h"
#include "Core/RealConstants.h"

#include "DataBase/DataBase.h"
#include "DataBase/DataBaseStruct.h"

#include "Definitions/MemoryAccessPattern.h"
#include "Definitions/PassiveScalar.h"

#include "FlowStateData/FlowStateData.cuh"
#include "FlowStateData/AccessDeviceData.cuh"

#include "CudaUtility/CudaRunKernel.hpp"

//////////////////////////////////////////////////////////////////////////

__global__                 void boundaryConditionKernel  ( const DataBaseStruct dataBase, 
                                                           const PressureStruct boundaryCondition, 
                                                           const Parameters parameters,
                                                           const uint startIndex,
                                                           const uint numberOfEntities );

__host__ __device__ inline void boundaryConditionFunction( const DataBaseStruct& dataBase, 
                                                           const PressureStruct& boundaryCondition, 
                                                           const Parameters& parameters,
                                                           const uint startIndex,
                                                           const uint index );

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

void Pressure::runBoundaryConditionKernel(const SPtr<DataBase> dataBase, 
                                                const Parameters parameters, 
                                                const uint level)
{    
    CudaUtility::CudaGrid grid( this->numberOfCellsPerLevel[ level ], 32 );

    runKernel( boundaryConditionKernel,
               boundaryConditionFunction,
               dataBase->getDeviceType(), grid, 
               dataBase->toStruct(),
               this->toStruct(),
               parameters,
               this->startOfCellsPerLevel[ level ] );

    hipDeviceSynchronize();

    getLastCudaError("Pressure::runBoundaryConditionKernel( const SPtr<DataBase> dataBase, const Parameters parameters, const uint level )");
}

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

__global__ void boundaryConditionKernel(const DataBaseStruct dataBase, 
                                        const PressureStruct boundaryCondition, 
                                        const Parameters parameters,
                                        const uint startIndex,
                                        const uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index >= numberOfEntities ) return;

    boundaryConditionFunction( dataBase, boundaryCondition, parameters, startIndex, index );
}

__host__ __device__ inline void boundaryConditionFunction(const DataBaseStruct& dataBase, 
                                                          const PressureStruct& boundaryCondition, 
                                                          const Parameters& parameters,
                                                          const uint startIndex,
                                                          const uint index)
{
    uint ghostCellIdx  = boundaryCondition.ghostCells [ startIndex + index ];
    uint domainCellIdx = boundaryCondition.domainCells[ startIndex + index ];
    uint secondCellIdx = boundaryCondition.secondCells[ startIndex + index ];

    PrimitiveVariables ghostCellPrim;
    {
        PrimitiveVariables domainCellPrim;
        PrimitiveVariables secondCellPrim;

        {
            ConservedVariables domainCellData;
            readCellData( domainCellIdx, dataBase, domainCellData );
            domainCellPrim = toPrimitiveVariables( domainCellData, parameters.K );

            ConservedVariables secondCellData;
            if( secondCellIdx != INVALID_INDEX ){
                readCellData( secondCellIdx, dataBase, secondCellData );
                secondCellPrim = toPrimitiveVariables( secondCellData, parameters.K );
            }
        }

        ghostCellPrim.U      = two * domainCellPrim.U      - secondCellPrim.U;
        ghostCellPrim.V      = two * domainCellPrim.V      - secondCellPrim.V;
        ghostCellPrim.W      = two * domainCellPrim.W      - secondCellPrim.W;
        ghostCellPrim.lambda = two * domainCellPrim.lambda - secondCellPrim.lambda;
    #ifdef USE_PASSIVE_SCALAR
        ghostCellPrim.S_1    = two * domainCellPrim.S_1    - secondCellPrim.S_1;
        ghostCellPrim.S_2    = two * domainCellPrim.S_2    - secondCellPrim.S_2;
    #endif // USE_PASSIVE_SCALAR

    //    ghostCellPrim.U      = domainCellPrim.U     ;
    //    ghostCellPrim.V      = domainCellPrim.V     ;
    //    ghostCellPrim.W      = domainCellPrim.W     ;
    //    ghostCellPrim.lambda = domainCellPrim.lambda;
    //#ifdef USE_PASSIVE_SCALAR
    //    ghostCellPrim.S      = domainCellPrim.S     ;
    //#endif // USE_PASSIVE_SCALAR


        real rho0 = ( two * boundaryCondition.p0 * c1o2 * ( domainCellPrim.lambda + ghostCellPrim.lambda ) );

        ghostCellPrim.rho = two * rho0 - domainCellPrim.rho;
    }

    {
        ConservedVariables ghostCons = toConservedVariables( ghostCellPrim, parameters.K );

        writeCellData( ghostCellIdx, dataBase, ghostCons );
    }
}

Pressure::Pressure(SPtr<DataBase> dataBase, real p0)
    : BoundaryCondition( dataBase )
{
    this->p0 = p0;
}

bool Pressure::isWall()
{
    return false;
}

bool Pressure::secondCellsNeeded()
{
    return true;
}

