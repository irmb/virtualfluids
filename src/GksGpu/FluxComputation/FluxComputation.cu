#include "hip/hip_runtime.h"
#include "FluxComputation.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Core/PointerDefinitions.h"

#include "DataBase/DataBaseStruct.h"

#include "Definitions/PassiveScalar.h"

#include "FlowStateData/FlowStateData.cuh"

#include "FluxComputation/Moments.cuh"
#include "FluxComputation/Reconstruction.cuh"
#include "FluxComputation/Transformation.cuh"
#include "FluxComputation/ExpansionCoefficients.cuh"
#include "FluxComputation/AssembleFlux.cuh"
#include "FluxComputation/ApplyFlux.cuh"

#include "CudaUtility/CudaRunKernel.hpp"

__global__                 void fluxKernel  ( DataBaseStruct dataBase, Parameters parameters, char direction, uint startIndex, uint numberOfEntities );

__host__ __device__ inline void fluxFunction( DataBaseStruct dataBase, Parameters parameters, char direction, uint startIndex, uint index );

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void FluxComputation::run( SPtr<DataBase> dataBase, Parameters parameters, uint level )
{
    {
        CudaUtility::CudaGrid grid(dataBase->perLevelCount[level].numberOfFacesX, 128);

        runKernel(fluxKernel,
                  fluxFunction,
                  dataBase->getDeviceType(), grid,
                  dataBase->toStruct(),
                  parameters,
                  'x',
                  dataBase->perLevelCount[level].startOfFacesX);

        hipDeviceSynchronize();

        getLastCudaError("FluxComputation::run( SPtr<DataBase> dataBase, Parameters parameters, 'x', uint level )");
    }
    {
        CudaUtility::CudaGrid grid(dataBase->perLevelCount[level].numberOfFacesY, 128);

        runKernel(fluxKernel,
                  fluxFunction,
                  dataBase->getDeviceType(), grid,
                  dataBase->toStruct(),
                  parameters,
                  'y',
                  dataBase->perLevelCount[level].startOfFacesY);

        hipDeviceSynchronize();

        getLastCudaError("FluxComputation::run( SPtr<DataBase> dataBase, Parameters parameters, 'y', uint level )");
    }
    {
        CudaUtility::CudaGrid grid(dataBase->perLevelCount[level].numberOfFacesZ, 128);

        runKernel(fluxKernel,
                  fluxFunction,
                  dataBase->getDeviceType(), grid,
                  dataBase->toStruct(),
                  parameters,
                  'z',
                  dataBase->perLevelCount[level].startOfFacesZ);

        hipDeviceSynchronize();

        getLastCudaError("FluxComputation::run( SPtr<DataBase> dataBase, Parameters parameters, 'z', uint level )");
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void fluxKernel(DataBaseStruct dataBase, Parameters parameters, char direction, uint startIndex, uint numberOfEntities)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index >= numberOfEntities ) return;

    fluxFunction( dataBase, parameters, direction, startIndex, index );
}

__host__ __device__ inline void fluxFunction(DataBaseStruct dataBase, Parameters parameters, char direction, uint startIndex, uint index)
{
    uint faceIndex = startIndex + index;

    //////////////////////////////////////////////////////////////////////////

    PrimitiveVariables facePrim;

    //////////////////////////////////////////////////////////////////////////

    real ax[LENGTH_CELL_DATA];
    real ay[LENGTH_CELL_DATA];
    real az[LENGTH_CELL_DATA];
    real at[LENGTH_CELL_DATA];

#pragma unroll
    for( uint i = 0; i < LENGTH_CELL_DATA; i++ )
    { 
        ax[i] = zero; 
        ay[i] = zero; 
        az[i] = zero; 
        at[i] = zero;
    }

    //////////////////////////////////////////////////////////////////////////

    {
        ConservedVariables gradN, gradT1, gradT2;

        reconstructFiniteDifferences(faceIndex,
                                     dataBase,
                                     parameters,
                                     direction,
                                     gradN,
                                     gradT1,
                                     gradT2,
                                     facePrim);

        transformGlobalToLocal( gradN , direction );
        transformGlobalToLocal( gradT1, direction );
        transformGlobalToLocal( gradT2, direction );

        transformGlobalToLocal( facePrim, direction );

        computeExpansionCoefficients(facePrim, gradN , parameters.K, ax);
        computeExpansionCoefficients(facePrim, gradT1, parameters.K, ay);
        computeExpansionCoefficients(facePrim, gradT2, parameters.K, az);
    }

    //////////////////////////////////////////////////////////////////////////

    {
        ConservedVariables flux;

        ConservedVariables faceCons;

        {
            real momentU [ NUMBER_OF_MOMENTS ]; 
            real momentV [ NUMBER_OF_MOMENTS ]; 
            real momentW [ NUMBER_OF_MOMENTS ]; 
            real momentXi[ NUMBER_OF_MOMENTS ];

            computeMoments( facePrim, parameters.K, momentU, momentV, momentW, momentXi );

            Vec3 force = parameters.force;

            transformGlobalToLocal(force, direction);

            {
                ConservedVariables timeGrad;
                computeTimeDerivative( facePrim, 
                                       momentU, 
                                       momentV, 
                                       momentW, 
                                       momentXi, 
                                       ax, ay, az,
                                       force,
                                       timeGrad );

                computeExpansionCoefficients( facePrim, timeGrad, parameters.K, at );
            }
            {
                real timeCoefficients[4];
                computeTimeCoefficients( facePrim, parameters, timeCoefficients );

                real heatFlux;
                assembleFlux( facePrim, 
                              momentU, momentV, momentW, momentXi,
                              ax, ay, az, at, 
                              timeCoefficients, 
                              parameters,
                              force,
                              flux,
                              heatFlux );

                transformLocalToGlobal( flux, direction );
            }
        }

        //////////////////////////////////////////////////////////////////////////

        {
            uint negCellIdx = dataBase.faceToCell[ NEG_CELL(faceIndex, dataBase.numberOfFaces) ];
            uint posCellIdx = dataBase.faceToCell[ POS_CELL(faceIndex, dataBase.numberOfFaces) ];

            if( dataBase.cellIsWall[ negCellIdx ] || dataBase.cellIsWall[ posCellIdx ] )
            {
                flux.rho = zero;
            }

            applyFluxToNegCell(dataBase, negCellIdx, flux, direction, parameters.dt);
            applyFluxToPosCell(dataBase, posCellIdx, flux, direction, parameters.dt);
        }
    }
}
