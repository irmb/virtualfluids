#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/D3Q27.h"
#include "GPU/constant.h"

//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipDevice27(doubflo* DD, 
                                         int* k_Q, 
                                         doubflo* QQ,
                                         unsigned int sizeQ,
                                         doubflo om1, 
                                         unsigned int* neighborX,
                                         unsigned int* neighborY,
                                         unsigned int* neighborZ,
                                         unsigned int size_Mat, 
                                         bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo f_W    = (D.f[dirE   ])[ke   ];
      doubflo f_E    = (D.f[dirW   ])[kw   ];
      doubflo f_S    = (D.f[dirN   ])[kn   ];
      doubflo f_N    = (D.f[dirS   ])[ks   ];
      doubflo f_B    = (D.f[dirT   ])[kt   ];
      doubflo f_T    = (D.f[dirB   ])[kb   ];
      doubflo f_SW   = (D.f[dirNE  ])[kne  ];
      doubflo f_NE   = (D.f[dirSW  ])[ksw  ];
      doubflo f_NW   = (D.f[dirSE  ])[kse  ];
      doubflo f_SE   = (D.f[dirNW  ])[knw  ];
      doubflo f_BW   = (D.f[dirTE  ])[kte  ];
      doubflo f_TE   = (D.f[dirBW  ])[kbw  ];
      doubflo f_TW   = (D.f[dirBE  ])[kbe  ];
      doubflo f_BE   = (D.f[dirTW  ])[ktw  ];
      doubflo f_BS   = (D.f[dirTN  ])[ktn  ];
      doubflo f_TN   = (D.f[dirBS  ])[kbs  ];
      doubflo f_TS   = (D.f[dirBN  ])[kbn  ];
      doubflo f_BN   = (D.f[dirTS  ])[kts  ];
      doubflo f_BSW  = (D.f[dirTNE ])[ktne ];
      doubflo f_BNE  = (D.f[dirTSW ])[ktsw ];
      doubflo f_BNW  = (D.f[dirTSE ])[ktse ];
      doubflo f_BSE  = (D.f[dirTNW ])[ktnw ];
      doubflo f_TSW  = (D.f[dirBNE ])[kbne ];
      doubflo f_TNE  = (D.f[dirBSW ])[kbsw ];
      doubflo f_TNW  = (D.f[dirBSE ])[kbse ];
      doubflo f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  ((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W); 
         

      vx2    =   (-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S); 

      vx3    =   ((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B); 

      doubflo cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  doubflo fac = one;//c99o100;
	  doubflo VeloX = fac*vx1;
	  doubflo VeloY = fac*vx2;
	  doubflo VeloZ = fac*vx3;
	  bool x = false;
	  bool y = false;
	  bool z = false;

      q = q_dirE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = zero;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cu_sq); 
         (D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-feq*om1)/(one-om1)+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = zero;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cu_sq); 
         (D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-feq*om1)/(one-om1)+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
		 VeloY = zero;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cu_sq); 
         (D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-feq*om1)/(one-om1)+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
		 VeloY = zero;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cu_sq); 
         (D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-feq*om1)/(one-om1)+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = zero;
		 z = true;
         feq=c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cu_sq); 
         (D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-feq*om1)/(one-om1)+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = zero;
		 z = true;
         feq=c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cu_sq); 
         (D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-feq*om1)/(one-om1)+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
         feq=c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         (D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-feq*om1)/(one-om1)+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
         feq=c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         (D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-feq*om1)/(one-om1)+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
         feq=c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         (D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-feq*om1)/(one-om1)+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
         feq=c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         (D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-feq*om1)/(one-om1)+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         (D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-feq*om1)/(one-om1)+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         (D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-feq*om1)/(one-om1)+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         (D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-feq*om1)/(one-om1)+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         (D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-feq*om1)/(one-om1)+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         (D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-feq*om1)/(one-om1)+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         (D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-feq*om1)/(one-om1)+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         (D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-feq*om1)/(one-om1)+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         (D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-feq*om1)/(one-om1)+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-feq*om1)/(one-om1)+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         (D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-feq*om1)/(one-om1)+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-feq*om1)/(one-om1)+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         (D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-feq*om1)/(one-om1)+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-feq*om1)/(one-om1)+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         (D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-feq*om1)/(one-om1)+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-feq*om1)/(one-om1)+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         (D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-feq*om1)/(one-om1)+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipDeviceComp27(doubflo* DD, 
											 int* k_Q, 
											 doubflo* QQ,
											 unsigned int sizeQ,
											 doubflo om1, 
											 unsigned int* neighborX,
											 unsigned int* neighborY,
											 unsigned int* neighborZ,
											 unsigned int size_Mat, 
											 bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo f_W    = (D.f[dirE   ])[ke   ];
      doubflo f_E    = (D.f[dirW   ])[kw   ];
      doubflo f_S    = (D.f[dirN   ])[kn   ];
      doubflo f_N    = (D.f[dirS   ])[ks   ];
      doubflo f_B    = (D.f[dirT   ])[kt   ];
      doubflo f_T    = (D.f[dirB   ])[kb   ];
      doubflo f_SW   = (D.f[dirNE  ])[kne  ];
      doubflo f_NE   = (D.f[dirSW  ])[ksw  ];
      doubflo f_NW   = (D.f[dirSE  ])[kse  ];
      doubflo f_SE   = (D.f[dirNW  ])[knw  ];
      doubflo f_BW   = (D.f[dirTE  ])[kte  ];
      doubflo f_TE   = (D.f[dirBW  ])[kbw  ];
      doubflo f_TW   = (D.f[dirBE  ])[kbe  ];
      doubflo f_BE   = (D.f[dirTW  ])[ktw  ];
      doubflo f_BS   = (D.f[dirTN  ])[ktn  ];
      doubflo f_TN   = (D.f[dirBS  ])[kbs  ];
      doubflo f_TS   = (D.f[dirBN  ])[kbn  ];
      doubflo f_BN   = (D.f[dirTS  ])[kts  ];
      doubflo f_BSW  = (D.f[dirTNE ])[ktne ];
      doubflo f_BNE  = (D.f[dirTSW ])[ktsw ];
      doubflo f_BNW  = (D.f[dirTSE ])[ktse ];
      doubflo f_BSE  = (D.f[dirTNW ])[ktnw ];
      doubflo f_TSW  = (D.f[dirBNE ])[kbne ];
      doubflo f_TNE  = (D.f[dirBSW ])[kbsw ];
      doubflo f_TNW  = (D.f[dirBSE ])[kbse ];
      doubflo f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (one + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (one + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (one + drho); 

      doubflo cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (one + drho);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //Test
      //(D.f[dirZERO])[k]=c1o10;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  doubflo fac = one;//c99o100;
	  doubflo VeloX = fac*vx1;
	  doubflo VeloY = fac*vx2;
	  doubflo VeloZ = fac*vx3;
	  bool x = false;
	  bool y = false;
	  bool z = false;

      q = q_dirE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = zero;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2over27* (drho/*+three*( vx1        )*/+c9over2*( vx1        )*( vx1        ) * (one + drho)-cu_sq); 
         (D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-f_W+(f_E+f_W-two*feq*om1)/(one-om1))*c1o2+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q) - c2over27 * drho;
         //feq=c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cu_sq); 
         //(D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-feq*om1)/(one-om1)+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q);
         //(D.f[dirW])[kw]=zero;
      }

      q = q_dirW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = zero;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 x = true;
         feq=c2over27* (drho/*+three*(-vx1        )*/+c9over2*(-vx1        )*(-vx1        ) * (one + drho)-cu_sq); 
         (D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-f_E+(f_W+f_E-two*feq*om1)/(one-om1))*c1o2+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q) - c2over27 * drho;
         //feq=c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cu_sq); 
         //(D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-feq*om1)/(one-om1)+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q);
         //(D.f[dirE])[ke]=zero;
      }

      q = q_dirN[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
		 VeloY = zero;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2over27* (drho/*+three*(    vx2     )*/+c9over2*(     vx2    )*(     vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-f_S+(f_N+f_S-two*feq*om1)/(one-om1))*c1o2+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q) - c2over27 * drho;
         //feq=c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cu_sq); 
         //(D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-feq*om1)/(one-om1)+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q);
         //(D.f[dirS])[ks]=zero;
      }

      q = q_dirS[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
		 VeloY = zero;
	     VeloZ = fac*vx3;
		 y = true;
         feq=c2over27* (drho/*+three*(   -vx2     )*/+c9over2*(    -vx2    )*(    -vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-f_N+(f_S+f_N-two*feq*om1)/(one-om1))*c1o2+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q) - c2over27 * drho;
         //feq=c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cu_sq); 
         //(D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-feq*om1)/(one-om1)+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q);
         //(D.f[dirN])[kn]=zero;
      }

      q = q_dirT[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = zero;
		 z = true;
         feq=c2over27* (drho/*+three*(         vx3)*/+c9over2*(         vx3)*(         vx3) * (one + drho)-cu_sq); 
         (D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-f_B+(f_T+f_B-two*feq*om1)/(one-om1))*c1o2+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q) - c2over27 * drho;
         //feq=c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cu_sq); 
         //(D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-feq*om1)/(one-om1)+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q);
         //(D.f[dirB])[kb]=one;
      }

      q = q_dirB[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
		 VeloZ = zero;
		 z = true;
         feq=c2over27* (drho/*+three*(        -vx3)*/+c9over2*(        -vx3)*(        -vx3) * (one + drho)-cu_sq); 
         (D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-f_T+(f_B+f_T-two*feq*om1)/(one-om1))*c1o2+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q) - c2over27 * drho;
         //feq=c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cu_sq); 
         //(D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-feq*om1)/(one-om1)+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q);
         //(D.f[dirT])[kt]=zero;
      }

      q = q_dirNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
         feq=c1over54* (drho/*+three*( vx1+vx2    )*/+c9over2*( vx1+vx2    )*( vx1+vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-f_SW+(f_NE+f_SW-two*feq*om1)/(one-om1))*c1o2+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cu_sq); 
         //(D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-feq*om1)/(one-om1)+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q);
         //(D.f[dirSW])[ksw]=zero;
      }

      q = q_dirSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
         feq=c1over54* (drho/*+three*(-vx1-vx2    )*/+c9over2*(-vx1-vx2    )*(-vx1-vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-f_NE+(f_SW+f_NE-two*feq*om1)/(one-om1))*c1o2+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cu_sq); 
         //(D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-feq*om1)/(one-om1)+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);
         //(D.f[dirNE])[kne]=zero;
      }

      q = q_dirSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
         feq=c1over54* (drho/*+three*( vx1-vx2    )*/+c9over2*( vx1-vx2    )*( vx1-vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-f_NW+(f_SE+f_NW-two*feq*om1)/(one-om1))*c1o2+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cu_sq); 
         //(D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-feq*om1)/(one-om1)+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);
         //(D.f[dirNW])[knw]=zero;
      }

      q = q_dirNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
         feq=c1over54* (drho/*+three*(-vx1+vx2    )*/+c9over2*(-vx1+vx2    )*(-vx1+vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-f_SE+(f_NW+f_SE-two*feq*om1)/(one-om1))*c1o2+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cu_sq); 
         //(D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-feq*om1)/(one-om1)+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);
         //(D.f[dirSE])[kse]=zero;
      }

      q = q_dirTE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho/*+three*( vx1    +vx3)*/+c9over2*( vx1    +vx3)*( vx1    +vx3) * (one + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-f_BW+(f_TE+f_BW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cu_sq); 
         //(D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-feq*om1)/(one-om1)+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);
         //(D.f[dirBW])[kbw]=zero;
      }

      q = q_dirBW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho/*+three*(-vx1    -vx3)*/+c9over2*(-vx1    -vx3)*(-vx1    -vx3) * (one + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-f_TE+(f_BW+f_TE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cu_sq); 
         //(D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-feq*om1)/(one-om1)+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);
         //(D.f[dirTE])[kte]=zero;
      }

      q = q_dirBE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho/*+three*( vx1    -vx3)*/+c9over2*( vx1    -vx3)*( vx1    -vx3) * (one + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-f_TW+(f_BE+f_TW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cu_sq); 
         //(D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-feq*om1)/(one-om1)+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);
         //(D.f[dirTW])[ktw]=zero;
      }

      q = q_dirTW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho/*+three*(-vx1    +vx3)*/+c9over2*(-vx1    +vx3)*(-vx1    +vx3) * (one + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-f_BE+(f_TW+f_BE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cu_sq); 
         //(D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-feq*om1)/(one-om1)+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);
         //(D.f[dirBE])[kbe]=zero;
      }

      q = q_dirTN[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho/*+three*(     vx2+vx3)*/+c9over2*(     vx2+vx3)*(     vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-f_BS+(f_TN+f_BS-two*feq*om1)/(one-om1))*c1o2+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cu_sq); 
         //(D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-feq*om1)/(one-om1)+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);
         //(D.f[dirBS])[kbs]=zero;
      }

      q = q_dirBS[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho/*+three*(    -vx2-vx3)*/+c9over2*(    -vx2-vx3)*(    -vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-f_TN+(f_BS+f_TN-two*feq*om1)/(one-om1))*c1o2+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cu_sq); 
         //(D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-feq*om1)/(one-om1)+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q);
         //(D.f[dirTN])[ktn]=zero;
      }

      q = q_dirBN[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho/*+three*(     vx2-vx3)*/+c9over2*(     vx2-vx3)*(     vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-f_TS+(f_BN+f_TS-two*feq*om1)/(one-om1))*c1o2+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cu_sq); 
         //(D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-feq*om1)/(one-om1)+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);
         //(D.f[dirTS])[kts]=zero;
      }

      q = q_dirTS[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over54* (drho/*+three*(    -vx2+vx3)*/+c9over2*(    -vx2+vx3)*(    -vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-f_BN+(f_TS+f_BN-two*feq*om1)/(one-om1))*c1o2+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q) - c1over54 * drho;
         //feq=c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cu_sq); 
         //(D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-feq*om1)/(one-om1)+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q);
         //(D.f[dirBN])[kbn]=zero;
      }

      q = q_dirTNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho/*+three*( vx1+vx2+vx3)*/+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //feq=c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cu_sq); 
         //(D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-feq*om1)/(one-om1)+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[dirBSW])[kbsw]=zero;
      }

      q = q_dirBSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho/*+three*(-vx1-vx2-vx3)*/+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //feq=c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cu_sq); 
         //(D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-feq*om1)/(one-om1)+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[dirTNE])[ktne]=zero;
      }

      q = q_dirBNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho/*+three*( vx1+vx2-vx3)*/+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //feq=c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cu_sq); 
         //(D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-feq*om1)/(one-om1)+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[dirTSW])[ktsw]=zero;
      }

      q = q_dirTSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho/*+three*(-vx1-vx2+vx3)*/+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //feq=c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cu_sq); 
         //(D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-feq*om1)/(one-om1)+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[dirBNE])[kbne]=zero;
      }

      q = q_dirTSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho/*+three*( vx1-vx2+vx3)*/+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //feq=c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cu_sq); 
         //(D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-feq*om1)/(one-om1)+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);
         //(D.f[dirBNW])[kbnw]=zero;
      }

      q = q_dirBNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho/*+three*(-vx1+vx2-vx3)*/+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //feq=c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cu_sq); 
         //(D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-feq*om1)/(one-om1)+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);
         //(D.f[dirTSE])[ktse]=zero;
      }

      q = q_dirBSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho/*+three*( vx1-vx2-vx3)*/+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q) - c1over216 * drho;
         //feq=c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cu_sq); 
         //(D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-feq*om1)/(one-om1)+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);
         //(D.f[dirTNW])[ktnw]=zero;
      }

      q = q_dirTNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = fac*vx1;
	     VeloY = fac*vx2;
	     VeloZ = fac*vx3;
		 if (x == true) VeloX = zero;
		 if (y == true) VeloY = zero;
		 if (z == true) VeloZ = zero;
         feq=c1over216*(drho/*+three*(-vx1+vx2+vx3)*/+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q) - c1over216 * drho;
         //feq=c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cu_sq); 
         //(D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-feq*om1)/(one-om1)+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);
         //(D.f[dirBSE])[kbse]=zero;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipGeomDeviceComp27(doubflo* DD, 
												 int* k_Q, 
												 doubflo* QQ,
												 unsigned int sizeQ,
												 doubflo om1, 
												 doubflo* NormalX,
												 doubflo* NormalY,
												 doubflo* NormalZ,
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[dirE   *sizeQ];
      nx_dirW   = &NormalX[dirW   *sizeQ];
      nx_dirN   = &NormalX[dirN   *sizeQ];
      nx_dirS   = &NormalX[dirS   *sizeQ];
      nx_dirT   = &NormalX[dirT   *sizeQ];
      nx_dirB   = &NormalX[dirB   *sizeQ];
      nx_dirNE  = &NormalX[dirNE  *sizeQ];
      nx_dirSW  = &NormalX[dirSW  *sizeQ];
      nx_dirSE  = &NormalX[dirSE  *sizeQ];
      nx_dirNW  = &NormalX[dirNW  *sizeQ];
      nx_dirTE  = &NormalX[dirTE  *sizeQ];
      nx_dirBW  = &NormalX[dirBW  *sizeQ];
      nx_dirBE  = &NormalX[dirBE  *sizeQ];
      nx_dirTW  = &NormalX[dirTW  *sizeQ];
      nx_dirTN  = &NormalX[dirTN  *sizeQ];
      nx_dirBS  = &NormalX[dirBS  *sizeQ];
      nx_dirBN  = &NormalX[dirBN  *sizeQ];
      nx_dirTS  = &NormalX[dirTS  *sizeQ];
      nx_dirTNE = &NormalX[dirTNE *sizeQ];
      nx_dirTSW = &NormalX[dirTSW *sizeQ];
      nx_dirTSE = &NormalX[dirTSE *sizeQ];
      nx_dirTNW = &NormalX[dirTNW *sizeQ];
      nx_dirBNE = &NormalX[dirBNE *sizeQ];
      nx_dirBSW = &NormalX[dirBSW *sizeQ];
      nx_dirBSE = &NormalX[dirBSE *sizeQ];
      nx_dirBNW = &NormalX[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[dirE   *sizeQ];
      ny_dirW   = &NormalY[dirW   *sizeQ];
      ny_dirN   = &NormalY[dirN   *sizeQ];
      ny_dirS   = &NormalY[dirS   *sizeQ];
      ny_dirT   = &NormalY[dirT   *sizeQ];
      ny_dirB   = &NormalY[dirB   *sizeQ];
      ny_dirNE  = &NormalY[dirNE  *sizeQ];
      ny_dirSW  = &NormalY[dirSW  *sizeQ];
      ny_dirSE  = &NormalY[dirSE  *sizeQ];
      ny_dirNW  = &NormalY[dirNW  *sizeQ];
      ny_dirTE  = &NormalY[dirTE  *sizeQ];
      ny_dirBW  = &NormalY[dirBW  *sizeQ];
      ny_dirBE  = &NormalY[dirBE  *sizeQ];
      ny_dirTW  = &NormalY[dirTW  *sizeQ];
      ny_dirTN  = &NormalY[dirTN  *sizeQ];
      ny_dirBS  = &NormalY[dirBS  *sizeQ];
      ny_dirBN  = &NormalY[dirBN  *sizeQ];
      ny_dirTS  = &NormalY[dirTS  *sizeQ];
      ny_dirTNE = &NormalY[dirTNE *sizeQ];
      ny_dirTSW = &NormalY[dirTSW *sizeQ];
      ny_dirTSE = &NormalY[dirTSE *sizeQ];
      ny_dirTNW = &NormalY[dirTNW *sizeQ];
      ny_dirBNE = &NormalY[dirBNE *sizeQ];
      ny_dirBSW = &NormalY[dirBSW *sizeQ];
      ny_dirBSE = &NormalY[dirBSE *sizeQ];
      ny_dirBNW = &NormalY[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[dirE   *sizeQ];
      nz_dirW   = &NormalZ[dirW   *sizeQ];
      nz_dirN   = &NormalZ[dirN   *sizeQ];
      nz_dirS   = &NormalZ[dirS   *sizeQ];
      nz_dirT   = &NormalZ[dirT   *sizeQ];
      nz_dirB   = &NormalZ[dirB   *sizeQ];
      nz_dirNE  = &NormalZ[dirNE  *sizeQ];
      nz_dirSW  = &NormalZ[dirSW  *sizeQ];
      nz_dirSE  = &NormalZ[dirSE  *sizeQ];
      nz_dirNW  = &NormalZ[dirNW  *sizeQ];
      nz_dirTE  = &NormalZ[dirTE  *sizeQ];
      nz_dirBW  = &NormalZ[dirBW  *sizeQ];
      nz_dirBE  = &NormalZ[dirBE  *sizeQ];
      nz_dirTW  = &NormalZ[dirTW  *sizeQ];
      nz_dirTN  = &NormalZ[dirTN  *sizeQ];
      nz_dirBS  = &NormalZ[dirBS  *sizeQ];
      nz_dirBN  = &NormalZ[dirBN  *sizeQ];
      nz_dirTS  = &NormalZ[dirTS  *sizeQ];
      nz_dirTNE = &NormalZ[dirTNE *sizeQ];
      nz_dirTSW = &NormalZ[dirTSW *sizeQ];
      nz_dirTSE = &NormalZ[dirTSE *sizeQ];
      nz_dirTNW = &NormalZ[dirTNW *sizeQ];
      nz_dirBNE = &NormalZ[dirBNE *sizeQ];
      nz_dirBSW = &NormalZ[dirBSW *sizeQ];
      nz_dirBSE = &NormalZ[dirBSE *sizeQ];
      nz_dirBNW = &NormalZ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo f_W    = (D.f[dirE   ])[ke   ];
      doubflo f_E    = (D.f[dirW   ])[kw   ];
      doubflo f_S    = (D.f[dirN   ])[kn   ];
      doubflo f_N    = (D.f[dirS   ])[ks   ];
      doubflo f_B    = (D.f[dirT   ])[kt   ];
      doubflo f_T    = (D.f[dirB   ])[kb   ];
      doubflo f_SW   = (D.f[dirNE  ])[kne  ];
      doubflo f_NE   = (D.f[dirSW  ])[ksw  ];
      doubflo f_NW   = (D.f[dirSE  ])[kse  ];
      doubflo f_SE   = (D.f[dirNW  ])[knw  ];
      doubflo f_BW   = (D.f[dirTE  ])[kte  ];
      doubflo f_TE   = (D.f[dirBW  ])[kbw  ];
      doubflo f_TW   = (D.f[dirBE  ])[kbe  ];
      doubflo f_BE   = (D.f[dirTW  ])[ktw  ];
      doubflo f_BS   = (D.f[dirTN  ])[ktn  ];
      doubflo f_TN   = (D.f[dirBS  ])[kbs  ];
      doubflo f_TS   = (D.f[dirBN  ])[kbn  ];
      doubflo f_BN   = (D.f[dirTS  ])[kts  ];
      doubflo f_BSW  = (D.f[dirTNE ])[ktne ];
      doubflo f_BNE  = (D.f[dirTSW ])[ktsw ];
      doubflo f_BNW  = (D.f[dirTSE ])[ktse ];
      doubflo f_BSE  = (D.f[dirTNW ])[ktnw ];
      doubflo f_TSW  = (D.f[dirBNE ])[kbne ];
      doubflo f_TNE  = (D.f[dirBSW ])[kbsw ];
      doubflo f_TNW  = (D.f[dirBSE ])[kbse ];
      doubflo f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (one + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (one + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (one + drho); 

      doubflo cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (one + drho);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  doubflo VeloX = vx1;
	  doubflo VeloY = vx2;
	  doubflo VeloZ = vx3;
	  doubflo fac = zero;//0.5;
 	  doubflo phi = zero;
	  doubflo alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      doubflo kxyFromfcNEQ = -(three * om1 / (one-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (one + drho) - ((vx1*vx2)));
      doubflo kyzFromfcNEQ = -(three * om1 / (one-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (one + drho) - ((vx2*vx3)));
      doubflo kxzFromfcNEQ = -(three * om1 / (one-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (one + drho) - ((vx1*vx3)));

	  doubflo kxxFromfcNEQ = -(three * om1 / (one-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (one + drho) - ((c1o3*drho + vx1*vx1)));
	  doubflo kyyFromfcNEQ = -(three * om1 / (one-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (one + drho) - ((c1o3*drho + vx2*vx2)));
	  doubflo kzzFromfcNEQ = -(three * om1 / (one-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (one + drho) - ((c1o3*drho + vx3*vx3)));

	  doubflo magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

	  //fac = fac * magS / (c1o3 * (one / om1 - c1o2));
   //   ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //doubflo *facAst = &QQ[dirZERO *sizeQ];

	  //fac = fac * alpha + facAst[k] * (one - alpha);
	  //facAst[k] = fac;
	  //(&QQ[dirZERO *sizeQ])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////doubflo uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //doubflo phi = expf(magS/0.01f) - one;
	  //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //doubflo C = five;
	  //doubflo kappa = 0.41f;
	  //doubflo phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
	  //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //doubflo sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
	  //doubflo qMed = sum/count;
	  //doubflo phi = fac / (qMed + fac);
	  //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  doubflo testQ = two;

      q = q_dirE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirE[k]) + fac);
		 VeloX *= phi;
         feq=c2over27* (drho/*+three*( vx1        )*/+c9over2*( vx1        )*( vx1        ) * (one + drho)-cu_sq); 
         (D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-f_W+(f_E+f_W-two*feq*om1)/(one-om1))*c1o2+(q*(f_E+f_W)-six*c2over27*( VeloX     ))/(one+q) - c2over27 * drho;
      }

      q = q_dirW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirW[k]) + fac);
		 VeloX *= phi;
         feq=c2over27* (drho/*+three*(-vx1        )*/+c9over2*(-vx1        )*(-vx1        ) * (one + drho)-cu_sq); 
         (D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-f_E+(f_W+f_E-two*feq*om1)/(one-om1))*c1o2+(q*(f_W+f_E)-six*c2over27*(-VeloX     ))/(one+q) - c2over27 * drho;
      }

      q = q_dirN[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirN[k]) + fac);
		 VeloY *= phi;
         feq=c2over27* (drho/*+three*(    vx2     )*/+c9over2*(     vx2    )*(     vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-f_S+(f_N+f_S-two*feq*om1)/(one-om1))*c1o2+(q*(f_N+f_S)-six*c2over27*( VeloY     ))/(one+q) - c2over27 * drho;
      }

      q = q_dirS[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirS[k]) + fac);
		 VeloY *= phi;
         feq=c2over27* (drho/*+three*(   -vx2     )*/+c9over2*(    -vx2    )*(    -vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-f_N+(f_S+f_N-two*feq*om1)/(one-om1))*c1o2+(q*(f_S+f_N)-six*c2over27*(-VeloY     ))/(one+q) - c2over27 * drho;
      }

      q = q_dirT[k];
      if (q>=zero && q<=one)
      {
		 VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nz_dirT[k]) + fac);
		 VeloZ *= phi;
         feq=c2over27* (drho/*+three*(         vx3)*/+c9over2*(         vx3)*(         vx3) * (one + drho)-cu_sq); 
         (D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-f_B+(f_T+f_B-two*feq*om1)/(one-om1))*c1o2+(q*(f_T+f_B)-six*c2over27*( VeloZ     ))/(one+q) - c2over27 * drho;
      }

      q = q_dirB[k];
      if (q>=zero && q<=one)
      {
		 VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nz_dirB[k]) + fac);
		 VeloZ *= phi;
         feq=c2over27* (drho/*+three*(        -vx3)*/+c9over2*(        -vx3)*(        -vx3) * (one + drho)-cu_sq); 
         (D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-f_T+(f_B+f_T-two*feq*om1)/(one-om1))*c1o2+(q*(f_B+f_T)-six*c2over27*(-VeloZ     ))/(one+q) - c2over27 * drho;
      }

      q = q_dirNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
		 VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirNE[k] + ny_dirNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1over54* (drho/*+three*( vx1+vx2    )*/+c9over2*( vx1+vx2    )*( vx1+vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-f_SW+(f_NE+f_SW-two*feq*om1)/(one-om1))*c1o2+(q*(f_NE+f_SW)-six*c1over54*(VeloX+VeloY))/(one+q) - c1over54 * drho;
      }

      q = q_dirSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
		 VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirSW[k] - ny_dirSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1over54* (drho/*+three*(-vx1-vx2    )*/+c9over2*(-vx1-vx2    )*(-vx1-vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-f_NE+(f_SW+f_NE-two*feq*om1)/(one-om1))*c1o2+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q) - c1over54 * drho;
      }

      q = q_dirSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
		 VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirSE[k] - ny_dirSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1over54* (drho/*+three*( vx1-vx2    )*/+c9over2*( vx1-vx2    )*( vx1-vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-f_NW+(f_SE+f_NW-two*feq*om1)/(one-om1))*c1o2+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q) - c1over54 * drho;
      }

      q = q_dirNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
		 VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirNW[k] + ny_dirNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
         feq=c1over54* (drho/*+three*(-vx1+vx2    )*/+c9over2*(-vx1+vx2    )*(-vx1+vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-f_SE+(f_NW+f_SE-two*feq*om1)/(one-om1))*c1o2+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q) - c1over54 * drho;
      }

      q = q_dirTE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTE[k] + nz_dirTE[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1over54* (drho/*+three*( vx1    +vx3)*/+c9over2*( vx1    +vx3)*( vx1    +vx3) * (one + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-f_BW+(f_TE+f_BW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q) - c1over54 * drho;
      }

      q = q_dirBW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBW[k] - nz_dirBW[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1over54* (drho/*+three*(-vx1    -vx3)*/+c9over2*(-vx1    -vx3)*(-vx1    -vx3) * (one + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-f_TE+(f_BW+f_TE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q) - c1over54 * drho;
      }

      q = q_dirBE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBE[k] - nz_dirBE[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1over54* (drho/*+three*( vx1    -vx3)*/+c9over2*( vx1    -vx3)*( vx1    -vx3) * (one + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-f_TW+(f_BE+f_TW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q) - c1over54 * drho;
      }

      q = q_dirTW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTW[k] + nz_dirTW[k]) + fac);
		 VeloX *= phi;
		 VeloZ *= phi;
         feq=c1over54* (drho/*+three*(-vx1    +vx3)*/+c9over2*(-vx1    +vx3)*(-vx1    +vx3) * (one + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-f_BE+(f_TW+f_BE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q) - c1over54 * drho;
      }

      q = q_dirTN[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
		 VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirTN[k] + nz_dirTN[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over54* (drho/*+three*(     vx2+vx3)*/+c9over2*(     vx2+vx3)*(     vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-f_BS+(f_TN+f_BS-two*feq*om1)/(one-om1))*c1o2+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q) - c1over54 * drho;
      }

      q = q_dirBS[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
		 VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirBS[k] - nz_dirBS[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over54* (drho/*+three*(    -vx2-vx3)*/+c9over2*(    -vx2-vx3)*(    -vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-f_TN+(f_BS+f_TN-two*feq*om1)/(one-om1))*c1o2+(q*(f_BS+f_TN)-six*c1over54*( -VeloY-VeloZ))/(one+q) - c1over54 * drho;
      }

      q = q_dirBN[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
		 VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( ny_dirBN[k] - nz_dirBN[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over54* (drho/*+three*(     vx2-vx3)*/+c9over2*(     vx2-vx3)*(     vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-f_TS+(f_BN+f_TS-two*feq*om1)/(one-om1))*c1o2+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q) - c1over54 * drho;
      }

      q = q_dirTS[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
		 VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-ny_dirTS[k] + nz_dirTS[k]) + fac);
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over54* (drho/*+three*(    -vx2+vx3)*/+c9over2*(    -vx2+vx3)*(    -vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-f_BN+(f_TS+f_BN-two*feq*om1)/(one-om1))*c1o2+(q*(f_TS+f_BN)-six*c1over54*( -VeloY+VeloZ))/(one+q) - c1over54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
		 VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over216*(drho/*+three*( vx1+vx2+vx3)*/+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q) - c1over216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
		 VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over216*(drho/*+three*(-vx1-vx2-vx3)*/+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q) - c1over216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
		 VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over216*(drho/*+three*( vx1+vx2-vx3)*/+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q) - c1over216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
		 VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over216*(drho/*+three*(-vx1-vx2+vx3)*/+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q) - c1over216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
		 VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over216*(drho/*+three*( vx1-vx2+vx3)*/+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q) - c1over216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
		 VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over216*(drho/*+three*(-vx1+vx2-vx3)*/+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q) - c1over216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
		 VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = (phi > one) ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over216*(drho/*+three*( vx1-vx2-vx3)*/+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q) - c1over216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
		 VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
		 //phi = fac * (one + magS / (Op0000002+uk) * (one-q));
		 //phi = phi > one ? one:phi;
		 //phi = fac; //Test
		 q = testQ; //AAAAHHHHHH bitte wieder raus nehmen!!!!
		 phi = fac / (q * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]) + fac);
		 VeloX *= phi;
		 VeloY *= phi;
		 VeloZ *= phi;
         feq=c1over216*(drho/*+three*(-vx1+vx2+vx3)*/+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q) - c1over216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QSlipNormDeviceComp27(doubflo* DD, 
												 int* k_Q, 
												 doubflo* QQ,
												 unsigned int sizeQ,
												 doubflo om1, 
												 doubflo* NormalX,
												 doubflo* NormalY,
												 doubflo* NormalZ,
												 unsigned int* neighborX,
												 unsigned int* neighborY,
												 unsigned int* neighborZ,
												 unsigned int size_Mat, 
												 bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<sizeQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *nx_dirE,   *nx_dirW,   *nx_dirN,   *nx_dirS,   *nx_dirT,   *nx_dirB, 
              *nx_dirNE,  *nx_dirSW,  *nx_dirSE,  *nx_dirNW,  *nx_dirTE,  *nx_dirBW,
              *nx_dirBE,  *nx_dirTW,  *nx_dirTN,  *nx_dirBS,  *nx_dirBN,  *nx_dirTS,
              *nx_dirTNE, *nx_dirTSW, *nx_dirTSE, *nx_dirTNW, *nx_dirBNE, *nx_dirBSW,
              *nx_dirBSE, *nx_dirBNW; 
      nx_dirE   = &NormalX[dirE   *sizeQ];
      nx_dirW   = &NormalX[dirW   *sizeQ];
      nx_dirN   = &NormalX[dirN   *sizeQ];
      nx_dirS   = &NormalX[dirS   *sizeQ];
      nx_dirT   = &NormalX[dirT   *sizeQ];
      nx_dirB   = &NormalX[dirB   *sizeQ];
      nx_dirNE  = &NormalX[dirNE  *sizeQ];
      nx_dirSW  = &NormalX[dirSW  *sizeQ];
      nx_dirSE  = &NormalX[dirSE  *sizeQ];
      nx_dirNW  = &NormalX[dirNW  *sizeQ];
      nx_dirTE  = &NormalX[dirTE  *sizeQ];
      nx_dirBW  = &NormalX[dirBW  *sizeQ];
      nx_dirBE  = &NormalX[dirBE  *sizeQ];
      nx_dirTW  = &NormalX[dirTW  *sizeQ];
      nx_dirTN  = &NormalX[dirTN  *sizeQ];
      nx_dirBS  = &NormalX[dirBS  *sizeQ];
      nx_dirBN  = &NormalX[dirBN  *sizeQ];
      nx_dirTS  = &NormalX[dirTS  *sizeQ];
      nx_dirTNE = &NormalX[dirTNE *sizeQ];
      nx_dirTSW = &NormalX[dirTSW *sizeQ];
      nx_dirTSE = &NormalX[dirTSE *sizeQ];
      nx_dirTNW = &NormalX[dirTNW *sizeQ];
      nx_dirBNE = &NormalX[dirBNE *sizeQ];
      nx_dirBSW = &NormalX[dirBSW *sizeQ];
      nx_dirBSE = &NormalX[dirBSE *sizeQ];
      nx_dirBNW = &NormalX[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *ny_dirE,   *ny_dirW,   *ny_dirN,   *ny_dirS,   *ny_dirT,   *ny_dirB, 
              *ny_dirNE,  *ny_dirSW,  *ny_dirSE,  *ny_dirNW,  *ny_dirTE,  *ny_dirBW,
              *ny_dirBE,  *ny_dirTW,  *ny_dirTN,  *ny_dirBS,  *ny_dirBN,  *ny_dirTS,
              *ny_dirTNE, *ny_dirTSW, *ny_dirTSE, *ny_dirTNW, *ny_dirBNE, *ny_dirBSW,
              *ny_dirBSE, *ny_dirBNW; 
      ny_dirE   = &NormalY[dirE   *sizeQ];
      ny_dirW   = &NormalY[dirW   *sizeQ];
      ny_dirN   = &NormalY[dirN   *sizeQ];
      ny_dirS   = &NormalY[dirS   *sizeQ];
      ny_dirT   = &NormalY[dirT   *sizeQ];
      ny_dirB   = &NormalY[dirB   *sizeQ];
      ny_dirNE  = &NormalY[dirNE  *sizeQ];
      ny_dirSW  = &NormalY[dirSW  *sizeQ];
      ny_dirSE  = &NormalY[dirSE  *sizeQ];
      ny_dirNW  = &NormalY[dirNW  *sizeQ];
      ny_dirTE  = &NormalY[dirTE  *sizeQ];
      ny_dirBW  = &NormalY[dirBW  *sizeQ];
      ny_dirBE  = &NormalY[dirBE  *sizeQ];
      ny_dirTW  = &NormalY[dirTW  *sizeQ];
      ny_dirTN  = &NormalY[dirTN  *sizeQ];
      ny_dirBS  = &NormalY[dirBS  *sizeQ];
      ny_dirBN  = &NormalY[dirBN  *sizeQ];
      ny_dirTS  = &NormalY[dirTS  *sizeQ];
      ny_dirTNE = &NormalY[dirTNE *sizeQ];
      ny_dirTSW = &NormalY[dirTSW *sizeQ];
      ny_dirTSE = &NormalY[dirTSE *sizeQ];
      ny_dirTNW = &NormalY[dirTNW *sizeQ];
      ny_dirBNE = &NormalY[dirBNE *sizeQ];
      ny_dirBSW = &NormalY[dirBSW *sizeQ];
      ny_dirBSE = &NormalY[dirBSE *sizeQ];
      ny_dirBNW = &NormalY[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo *nz_dirE,   *nz_dirW,   *nz_dirN,   *nz_dirS,   *nz_dirT,   *nz_dirB, 
              *nz_dirNE,  *nz_dirSW,  *nz_dirSE,  *nz_dirNW,  *nz_dirTE,  *nz_dirBW,
              *nz_dirBE,  *nz_dirTW,  *nz_dirTN,  *nz_dirBS,  *nz_dirBN,  *nz_dirTS,
              *nz_dirTNE, *nz_dirTSW, *nz_dirTSE, *nz_dirTNW, *nz_dirBNE, *nz_dirBSW,
              *nz_dirBSE, *nz_dirBNW; 
      nz_dirE   = &NormalZ[dirE   *sizeQ];
      nz_dirW   = &NormalZ[dirW   *sizeQ];
      nz_dirN   = &NormalZ[dirN   *sizeQ];
      nz_dirS   = &NormalZ[dirS   *sizeQ];
      nz_dirT   = &NormalZ[dirT   *sizeQ];
      nz_dirB   = &NormalZ[dirB   *sizeQ];
      nz_dirNE  = &NormalZ[dirNE  *sizeQ];
      nz_dirSW  = &NormalZ[dirSW  *sizeQ];
      nz_dirSE  = &NormalZ[dirSE  *sizeQ];
      nz_dirNW  = &NormalZ[dirNW  *sizeQ];
      nz_dirTE  = &NormalZ[dirTE  *sizeQ];
      nz_dirBW  = &NormalZ[dirBW  *sizeQ];
      nz_dirBE  = &NormalZ[dirBE  *sizeQ];
      nz_dirTW  = &NormalZ[dirTW  *sizeQ];
      nz_dirTN  = &NormalZ[dirTN  *sizeQ];
      nz_dirBS  = &NormalZ[dirBS  *sizeQ];
      nz_dirBN  = &NormalZ[dirBN  *sizeQ];
      nz_dirTS  = &NormalZ[dirTS  *sizeQ];
      nz_dirTNE = &NormalZ[dirTNE *sizeQ];
      nz_dirTSW = &NormalZ[dirTSW *sizeQ];
      nz_dirTSE = &NormalZ[dirTSE *sizeQ];
      nz_dirTNW = &NormalZ[dirTNW *sizeQ];
      nz_dirBNE = &NormalZ[dirBNE *sizeQ];
      nz_dirBSW = &NormalZ[dirBSW *sizeQ];
      nz_dirBSE = &NormalZ[dirBSE *sizeQ];
      nz_dirBNW = &NormalZ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KQK  = k_Q[k];
      unsigned int kzero= KQK;
      unsigned int ke   = KQK;
      unsigned int kw   = neighborX[KQK];
      unsigned int kn   = KQK;
      unsigned int ks   = neighborY[KQK];
      unsigned int kt   = KQK;
      unsigned int kb   = neighborZ[KQK];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = KQK;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = KQK;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = KQK;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = KQK;
      unsigned int kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo f_W    = (D.f[dirE   ])[ke   ];
      doubflo f_E    = (D.f[dirW   ])[kw   ];
      doubflo f_S    = (D.f[dirN   ])[kn   ];
      doubflo f_N    = (D.f[dirS   ])[ks   ];
      doubflo f_B    = (D.f[dirT   ])[kt   ];
      doubflo f_T    = (D.f[dirB   ])[kb   ];
      doubflo f_SW   = (D.f[dirNE  ])[kne  ];
      doubflo f_NE   = (D.f[dirSW  ])[ksw  ];
      doubflo f_NW   = (D.f[dirSE  ])[kse  ];
      doubflo f_SE   = (D.f[dirNW  ])[knw  ];
      doubflo f_BW   = (D.f[dirTE  ])[kte  ];
      doubflo f_TE   = (D.f[dirBW  ])[kbw  ];
      doubflo f_TW   = (D.f[dirBE  ])[kbe  ];
      doubflo f_BE   = (D.f[dirTW  ])[ktw  ];
      doubflo f_BS   = (D.f[dirTN  ])[ktn  ];
      doubflo f_TN   = (D.f[dirBS  ])[kbs  ];
      doubflo f_TS   = (D.f[dirBN  ])[kbn  ];
      doubflo f_BN   = (D.f[dirTS  ])[kts  ];
      doubflo f_BSW  = (D.f[dirTNE ])[ktne ];
      doubflo f_BNE  = (D.f[dirTSW ])[ktsw ];
      doubflo f_BNW  = (D.f[dirTSE ])[ktse ];
      doubflo f_BSE  = (D.f[dirTNW ])[ktnw ];
      doubflo f_TSW  = (D.f[dirBNE ])[kbne ];
      doubflo f_TNE  = (D.f[dirBSW ])[kbsw ];
      doubflo f_TNW  = (D.f[dirBSE ])[kbse ];
      doubflo f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      doubflo vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (one + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (one + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (one + drho); 

      doubflo cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (one + drho);

      //////////////////////////////////////////////////////////////////////////
      if (evenOrOdd==false)
      {
         D.f[dirE   ] = &DD[dirE   *size_Mat];
         D.f[dirW   ] = &DD[dirW   *size_Mat];
         D.f[dirN   ] = &DD[dirN   *size_Mat];
         D.f[dirS   ] = &DD[dirS   *size_Mat];
         D.f[dirT   ] = &DD[dirT   *size_Mat];
         D.f[dirB   ] = &DD[dirB   *size_Mat];
         D.f[dirNE  ] = &DD[dirNE  *size_Mat];
         D.f[dirSW  ] = &DD[dirSW  *size_Mat];
         D.f[dirSE  ] = &DD[dirSE  *size_Mat];
         D.f[dirNW  ] = &DD[dirNW  *size_Mat];
         D.f[dirTE  ] = &DD[dirTE  *size_Mat];
         D.f[dirBW  ] = &DD[dirBW  *size_Mat];
         D.f[dirBE  ] = &DD[dirBE  *size_Mat];
         D.f[dirTW  ] = &DD[dirTW  *size_Mat];
         D.f[dirTN  ] = &DD[dirTN  *size_Mat];
         D.f[dirBS  ] = &DD[dirBS  *size_Mat];
         D.f[dirBN  ] = &DD[dirBN  *size_Mat];
         D.f[dirTS  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirTNE *size_Mat];
         D.f[dirTSW ] = &DD[dirTSW *size_Mat];
         D.f[dirTSE ] = &DD[dirTSE *size_Mat];
         D.f[dirTNW ] = &DD[dirTNW *size_Mat];
         D.f[dirBNE ] = &DD[dirBNE *size_Mat];
         D.f[dirBSW ] = &DD[dirBSW *size_Mat];
         D.f[dirBSE ] = &DD[dirBSE *size_Mat];
         D.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         D.f[dirW   ] = &DD[dirE   *size_Mat];
         D.f[dirE   ] = &DD[dirW   *size_Mat];
         D.f[dirS   ] = &DD[dirN   *size_Mat];
         D.f[dirN   ] = &DD[dirS   *size_Mat];
         D.f[dirB   ] = &DD[dirT   *size_Mat];
         D.f[dirT   ] = &DD[dirB   *size_Mat];
         D.f[dirSW  ] = &DD[dirNE  *size_Mat];
         D.f[dirNE  ] = &DD[dirSW  *size_Mat];
         D.f[dirNW  ] = &DD[dirSE  *size_Mat];
         D.f[dirSE  ] = &DD[dirNW  *size_Mat];
         D.f[dirBW  ] = &DD[dirTE  *size_Mat];
         D.f[dirTE  ] = &DD[dirBW  *size_Mat];
         D.f[dirTW  ] = &DD[dirBE  *size_Mat];
         D.f[dirBE  ] = &DD[dirTW  *size_Mat];
         D.f[dirBS  ] = &DD[dirTN  *size_Mat];
         D.f[dirTN  ] = &DD[dirBS  *size_Mat];
         D.f[dirTS  ] = &DD[dirBN  *size_Mat];
         D.f[dirBN  ] = &DD[dirTS  *size_Mat];
         D.f[dirZERO] = &DD[dirZERO*size_Mat];
         D.f[dirTNE ] = &DD[dirBSW *size_Mat];
         D.f[dirTSW ] = &DD[dirBNE *size_Mat];
         D.f[dirTSE ] = &DD[dirBNW *size_Mat];
         D.f[dirTNW ] = &DD[dirBSE *size_Mat];
         D.f[dirBNE ] = &DD[dirTSW *size_Mat];
         D.f[dirBSW ] = &DD[dirTNE *size_Mat];
         D.f[dirBSE ] = &DD[dirTNW *size_Mat];
         D.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  doubflo VeloX = vx1;
	  doubflo VeloY = vx2;
	  doubflo VeloZ = vx3;
	  doubflo fac = c1o100;//0.5;
 	  doubflo phi = zero;
	  doubflo alpha = c1o100;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      doubflo kxyFromfcNEQ = -(three * om1 / (one-om1))*((f_SW+f_BSW+f_TSW-f_NW-f_BNW-f_TNW-f_SE-f_BSE-f_TSE+f_NE+f_BNE+f_TNE ) / (one + drho) - ((vx1*vx2)));
      doubflo kyzFromfcNEQ = -(three * om1 / (one-om1))*((f_BS+f_BSE+f_BSW-f_TS-f_TSE-f_TSW-f_BN-f_BNE-f_BNW+f_TN+f_TNE+f_TNW ) / (one + drho) - ((vx2*vx3)));
      doubflo kxzFromfcNEQ = -(three * om1 / (one-om1))*((f_BW+f_BSW+f_BNW-f_TW-f_TSW-f_TNW-f_BE-f_BSE-f_BNE+f_TE+f_TSE+f_TNE ) / (one + drho) - ((vx1*vx3)));

	  doubflo kxxFromfcNEQ = -(three * om1 / (one-om1))*((f_E+f_NE+f_SE+f_TE+f_BE+f_W+f_NW+f_SW+f_TW+f_BW+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (one + drho) - ((c1o3*drho + vx1*vx1)));
	  doubflo kyyFromfcNEQ = -(three * om1 / (one-om1))*((f_N+f_NE+f_NW+f_TN+f_BN+f_S+f_SE+f_SW+f_TS+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (one + drho) - ((c1o3*drho + vx2*vx2)));
	  doubflo kzzFromfcNEQ = -(three * om1 / (one-om1))*((f_T+f_TE+f_TW+f_TN+f_BS+f_B+f_BE+f_BW+f_BN+f_BS+ f_TNE+f_TSE+f_BNE+f_TNE+ f_TNW+f_TSW+f_BNW+f_TNW ) / (one + drho) - ((c1o3*drho + vx3*vx3)));

	  doubflo magS = sqrtf(kxyFromfcNEQ*kxyFromfcNEQ + kyzFromfcNEQ*kyzFromfcNEQ + kxzFromfcNEQ*kxzFromfcNEQ + kxxFromfcNEQ*kxxFromfcNEQ + kyyFromfcNEQ*kyyFromfcNEQ + kzzFromfcNEQ*kzzFromfcNEQ);

	  fac = fac * magS / (c1o3 * (one / om1 - c1o2));
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  doubflo *facAst = &QQ[dirZERO *sizeQ];

	  fac = fac * alpha + facAst[k] * (one - alpha);
	  facAst[k] = fac;
	  //(&QQ[dirZERO *sizeQ])[KQK] = fac;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  ////doubflo uk = sqrtf(vx1*vx1 + vx2*vx2 + vx3*vx3);
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //doubflo phi = expf(magS/0.01f) - one;
	  //phi = (phi > one) ? one:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //doubflo C = five;
	  //doubflo kappa = 0.41f;
	  //doubflo phi = (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))) - one) / (C * kappa * c1o2 * logf(magS / (c1o3 * (one / om1 - c1o2))));
	  //phi = (phi < zero) ? zero:phi;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //doubflo sum = zero, count = zero;
   //   q = q_dirE   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirE[k] ); count += one;
   //   q = q_dirW   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirW[k])); count += one;
   //   q = q_dirN   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirN[k] ); count += one;
   //   q = q_dirS   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirS[k])); count += one;
   //   q = q_dirT   [k]; if (q>=zero && q<=one) sum += (q *   nx_dirT[k] ); count += one;
   //   q = q_dirB   [k]; if (q>=zero && q<=one) sum += (q * (-nx_dirB[k])); count += one;
   //   q = q_dirNE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirNE[k]  + ny_dirNE[k])/(sqrtf(two))); count += one;
   //   q = q_dirSW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirSW[k]) - ny_dirSW[k])/(sqrtf(two))); count += one;
   //   q = q_dirSE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirSE[k]  - ny_dirSE[k])/(sqrtf(two))); count += one;
   //   q = q_dirNW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirNW[k]) + ny_dirNW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTE[k]  + nz_dirTE[k])/(sqrtf(two))); count += one;
   //   q = q_dirBW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBW[k]) - nz_dirBW[k])/(sqrtf(two))); count += one;
   //   q = q_dirBE  [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBE[k]  - nz_dirBE[k])/(sqrtf(two))); count += one;
   //   q = q_dirTW  [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTW[k]) + nz_dirTW[k])/(sqrtf(two))); count += one;
   //   q = q_dirTN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirTN[k]  + nz_dirTN[k])/(sqrtf(two))); count += one;
   //   q = q_dirBS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirBS[k]) - nz_dirBS[k])/(sqrtf(two))); count += one;
   //   q = q_dirBN  [k]; if (q>=zero && q<=one) sum += (q * (  ny_dirBN[k]  - nz_dirBN[k])/(sqrtf(two))); count += one;
   //   q = q_dirTS  [k]; if (q>=zero && q<=one) sum += (q * ((-ny_dirTS[k]) + nz_dirTS[k])/(sqrtf(two))); count += one;
   //   q = q_dirTNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTSW[k])- ny_dirTSW[k] + nz_dirTSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirTSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirTNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirTNW[k])+ ny_dirTNW[k] + nz_dirTNW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBSW[k])- ny_dirBSW[k] - nz_dirBSW[k])/(sqrtf(three))); count += one;
   //   q = q_dirBSE [k]; if (q>=zero && q<=one) sum += (q * (  nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k])/(sqrtf(three))); count += one;
   //   q = q_dirBNW [k]; if (q>=zero && q<=one) sum += (q * ((-nx_dirBNW[k])+ ny_dirBNW[k] - nz_dirBNW[k])/(sqrtf(three))); count += one;
	  //doubflo qMed = sum/count;
	  //doubflo phi = fac / (qMed + fac);
	  //phi = (phi > one) ? one:one;
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  doubflo sliplength = 0.9f;//c1o2;
	  doubflo qSlip = zero;
	  doubflo un = zero;
	  doubflo ut = zero;
	  doubflo tangential = zero;
	  //doubflo smallSingle = Op0000002;

      q = q_dirE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k];
		 un = fabs((vx1 * nx_dirE[k] + vx2 * ny_dirE[k] + vx3 * nz_dirE[k]) * nx_dirE[k]);
		 ut = fabs(VeloX);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c2over27* (drho/*+three*( vx1        )*/+c9over2*( vx1        )*( vx1        ) * (one + drho)-cu_sq); 
         (D.f[dirW])[kw]=(one-q)/(one+q)*(f_E-f_W+(f_E+f_W-two*feq*om1)/(one-om1))*c1o2+(q*(f_E+f_W))/(one+q) - c2over27 * drho;
      }

      q = q_dirW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k];
		 un = fabs(-(vx1 * nx_dirW[k] + vx2 * ny_dirW[k] + vx3 * nz_dirW[k]) * nx_dirW[k]);
		 ut = fabs(-VeloX);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c2over27* (drho/*+three*(-vx1        )*/+c9over2*(-vx1        )*(-vx1        ) * (one + drho)-cu_sq); 
         (D.f[dirE])[ke]=(one-q)/(one+q)*(f_W-f_E+(f_W+f_E-two*feq*om1)/(one-om1))*c1o2+(q*(f_W+f_E))/(one+q) - c2over27 * drho;
      }

      q = q_dirN[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k];
		 un = fabs( (vx1 * nx_dirN[k] + vx2 * ny_dirN[k] + vx3 * nz_dirN[k]) * ny_dirN[k]);
		 ut = fabs( VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c2over27* (drho/*+three*(    vx2     )*/+c9over2*(     vx2    )*(     vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirS])[ks]=(one-q)/(one+q)*(f_N-f_S+(f_N+f_S-two*feq*om1)/(one-om1))*c1o2+(q*(f_N+f_S))/(one+q) - c2over27 * drho;
      }

      q = q_dirS[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k];
		 un = fabs(-(vx1 * nx_dirS[k] + vx2 * ny_dirS[k] + vx3 * nz_dirS[k]) * ny_dirS[k]);
		 ut = fabs(-VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c2over27* (drho/*+three*(   -vx2     )*/+c9over2*(    -vx2    )*(    -vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirN])[kn]=(one-q)/(one+q)*(f_S-f_N+(f_S+f_N-two*feq*om1)/(one-om1))*c1o2+(q*(f_S+f_N))/(one+q) - c2over27 * drho;
      }

      q = q_dirT[k];
      if (q>=zero && q<=one)
      {
		 VeloZ = vx3 - (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k];
		 un = fabs( (vx1 * nx_dirT[k] + vx2 * ny_dirT[k] + vx3 * nz_dirT[k]) * nz_dirT[k]);
		 ut = fabs( VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nz_dirT[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c2over27* (drho/*+three*(         vx3)*/+c9over2*(         vx3)*(         vx3) * (one + drho)-cu_sq); 
         (D.f[dirB])[kb]=(one-q)/(one+q)*(f_T-f_B+(f_T+f_B-two*feq*om1)/(one-om1))*c1o2+(q*(f_T+f_B))/(one+q) - c2over27 * drho;
      }

      q = q_dirB[k];
      if (q>=zero && q<=one)
      {
		 VeloZ = vx3 - (vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k];
		 un = fabs(-(vx1 * nx_dirB[k] + vx2 * ny_dirB[k] + vx3 * nz_dirB[k]) * nz_dirB[k]);
		 ut = fabs(-VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nz_dirB[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c2over27* (drho/*+three*(        -vx3)*/+c9over2*(        -vx3)*(        -vx3) * (one + drho)-cu_sq); 
         (D.f[dirT])[kt]=(one-q)/(one+q)*(f_B-f_T+(f_B+f_T-two*feq*om1)/(one-om1))*c1o2+(q*(f_B+f_T))/(one+q) - c2over27 * drho;
      }

      q = q_dirNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k];
		 VeloY = vx2 - (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k];
		 un = fabs( (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * nx_dirNE[k] + (vx1 * nx_dirNE[k] + vx2 * ny_dirNE[k] + vx3 * nz_dirNE[k]) * ny_dirNE[k]);
		 ut = fabs( VeloX + VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirNE[k]+ny_dirNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*( vx1+vx2    )*/+c9over2*( vx1+vx2    )*( vx1+vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-f_SW+(f_NE+f_SW-two*feq*om1)/(one-om1))*c1o2+(q*(f_NE+f_SW))/(one+q) - c1over54 * drho;
      }

      q = q_dirSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k];
		 VeloY = vx2 - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k];
		 un = fabs(-(vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * nx_dirSW[k] - (vx1 * nx_dirSW[k] + vx2 * ny_dirSW[k] + vx3 * nz_dirSW[k]) * ny_dirSW[k]);
		 ut = fabs(-VeloX - VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirSW[k]-ny_dirSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*(-vx1-vx2    )*/+c9over2*(-vx1-vx2    )*(-vx1-vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-f_NE+(f_SW+f_NE-two*feq*om1)/(one-om1))*c1o2+(q*(f_SW+f_NE))/(one+q) - c1over54 * drho;
      }

      q = q_dirSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k];
		 VeloY = vx2 - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k];
		 un = fabs( (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * nx_dirSE[k] - (vx1 * nx_dirSE[k] + vx2 * ny_dirSE[k] + vx3 * nz_dirSE[k]) * ny_dirSE[k]);
		 ut = fabs( VeloX - VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirSE[k]-ny_dirSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*( vx1-vx2    )*/+c9over2*( vx1-vx2    )*( vx1-vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-f_NW+(f_SE+f_NW-two*feq*om1)/(one-om1))*c1o2+(q*(f_SE+f_NW))/(one+q) - c1over54 * drho;
      }

      q = q_dirNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k];
		 VeloY = vx2 - (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k];
		 un = fabs(-(vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * nx_dirNW[k] + (vx1 * nx_dirNW[k] + vx2 * ny_dirNW[k] + vx3 * nz_dirNW[k]) * ny_dirNW[k]);
		 ut = fabs(-VeloX + VeloY);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirNW[k]+ny_dirNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*(-vx1+vx2    )*/+c9over2*(-vx1+vx2    )*(-vx1+vx2    ) * (one + drho)-cu_sq); 
         (D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-f_SE+(f_NW+f_SE-two*feq*om1)/(one-om1))*c1o2+(q*(f_NW+f_SE))/(one+q) - c1over54 * drho;
      }

      q = q_dirTE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k];
		 un = fabs( (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nx_dirTE[k] + (vx1 * nx_dirTE[k] + vx2 * ny_dirTE[k] + vx3 * nz_dirTE[k]) * nz_dirTE[k]);
		 ut = fabs( VeloX + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTE[k]+nz_dirTE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*( vx1    +vx3)*/+c9over2*( vx1    +vx3)*( vx1    +vx3) * (one + drho)-cu_sq); 
         (D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-f_BW+(f_TE+f_BW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TE+f_BW))/(one+q) - c1over54 * drho;
      }

      q = q_dirBW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k];
		 un = fabs(-(vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nx_dirBW[k] - (vx1 * nx_dirBW[k] + vx2 * ny_dirBW[k] + vx3 * nz_dirBW[k]) * nz_dirBW[k]);
		 ut = fabs(-VeloX - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBW[k]-nz_dirBW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*(-vx1    -vx3)*/+c9over2*(-vx1    -vx3)*(-vx1    -vx3) * (one + drho)-cu_sq); 
         (D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-f_TE+(f_BW+f_TE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BW+f_TE))/(one+q) - c1over54 * drho;
      }

      q = q_dirBE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k];
		 un = fabs( (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nx_dirBE[k] - (vx1 * nx_dirBE[k] + vx2 * ny_dirBE[k] + vx3 * nz_dirBE[k]) * nz_dirBE[k]);
		 ut = fabs( VeloX - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBE[k]-nz_dirBE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*( vx1    -vx3)*/+c9over2*( vx1    -vx3)*( vx1    -vx3) * (one + drho)-cu_sq); 
         (D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-f_TW+(f_BE+f_TW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BE+f_TW))/(one+q) - c1over54 * drho;
      }

      q = q_dirTW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k];
		 un = fabs(-(vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nx_dirTW[k] + (vx1 * nx_dirTW[k] + vx2 * ny_dirTW[k] + vx3 * nz_dirTW[k]) * nz_dirTW[k]);
		 ut = fabs(-VeloX + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTW[k]+nz_dirTW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*(-vx1    +vx3)*/+c9over2*(-vx1    +vx3)*(-vx1    +vx3) * (one + drho)-cu_sq); 
         (D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-f_BE+(f_TW+f_BE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TW+f_BE))/(one+q) - c1over54 * drho;
      }

      q = q_dirTN[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k];
		 VeloZ = vx3 - (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k];
		 un = fabs( (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * ny_dirTN[k] + (vx1 * nx_dirTN[k] + vx2 * ny_dirTN[k] + vx3 * nz_dirTN[k]) * nz_dirTN[k]);
		 ut = fabs( VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirTN[k]+nz_dirTN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*(     vx2+vx3)*/+c9over2*(     vx2+vx3)*(     vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-f_BS+(f_TN+f_BS-two*feq*om1)/(one-om1))*c1o2+(q*(f_TN+f_BS))/(one+q) - c1over54 * drho;
      }

      q = q_dirBS[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k];
		 VeloZ = vx3 - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k];
		 un = fabs(-(vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * ny_dirBS[k] - (vx1 * nx_dirBS[k] + vx2 * ny_dirBS[k] + vx3 * nz_dirBS[k]) * nz_dirBS[k]);
		 ut = fabs(-VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirBS[k]-nz_dirBS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*(    -vx2-vx3)*/+c9over2*(    -vx2-vx3)*(    -vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-f_TN+(f_BS+f_TN-two*feq*om1)/(one-om1))*c1o2+(q*(f_BS+f_TN))/(one+q) - c1over54 * drho;
      }

      q = q_dirBN[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k];
		 VeloZ = vx3 - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k];
		 un = fabs( (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * ny_dirBN[k] - (vx1 * nx_dirBN[k] + vx2 * ny_dirBN[k] + vx3 * nz_dirBN[k]) * nz_dirBN[k]);
		 ut = fabs( VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( ny_dirBN[k]-nz_dirBN[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*(     vx2-vx3)*/+c9over2*(     vx2-vx3)*(     vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-f_TS+(f_BN+f_TS-two*feq*om1)/(one-om1))*c1o2+(q*(f_BN+f_TS))/(one+q) - c1over54 * drho;
      }

      q = q_dirTS[k];
      if (q>=zero && q<=one)
      {
		 VeloY = vx2 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k];
		 VeloZ = vx3 - (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k];
		 un = fabs(-(vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * ny_dirTS[k] + (vx1 * nx_dirTS[k] + vx2 * ny_dirTS[k] + vx3 * nz_dirTS[k]) * nz_dirTS[k]);
		 ut = fabs(-VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-ny_dirTS[k]+nz_dirTS[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over54* (drho/*+three*(    -vx2+vx3)*/+c9over2*(    -vx2+vx3)*(    -vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-f_BN+(f_TS+f_BN-two*feq*om1)/(one-om1))*c1o2+(q*(f_TS+f_BN))/(one+q) - c1over54 * drho;
      }

      q = q_dirTNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k];
		 VeloY = vx2 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k];
		 un = fabs( (vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nx_dirTNE[k] 
				   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * ny_dirTNE[k] 
				   +(vx1 * nx_dirTNE[k] + vx2 * ny_dirTNE[k] + vx3 * nz_dirTNE[k]) * nz_dirTNE[k]);
		 ut = fabs( VeloX + VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTNE[k] + ny_dirTNE[k] + nz_dirTNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over216*(drho/*+three*( vx1+vx2+vx3)*/+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNE+f_BSW))/(one+q) - c1over216 * drho;
      }

      q = q_dirBSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k];
		 VeloY = vx2 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k];
		 un = fabs(-(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nx_dirBSW[k] 
				   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * ny_dirBSW[k] 
				   -(vx1 * nx_dirBSW[k] + vx2 * ny_dirBSW[k] + vx3 * nz_dirBSW[k]) * nz_dirBSW[k]);
		 ut = fabs(-VeloX - VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBSW[k] - ny_dirBSW[k] - nz_dirBSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over216*(drho/*+three*(-vx1-vx2-vx3)*/+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSW+f_TNE))/(one+q) - c1over216 * drho;
      }

      q = q_dirBNE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k];
		 VeloY = vx2 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k];
		 un = fabs( (vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nx_dirBNE[k] 
				   +(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * ny_dirBNE[k] 
				   -(vx1 * nx_dirBNE[k] + vx2 * ny_dirBNE[k] + vx3 * nz_dirBNE[k]) * nz_dirBNE[k]);
		 ut = fabs( VeloX + VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBNE[k] + ny_dirBNE[k] - nz_dirBNE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over216*(drho/*+three*( vx1+vx2-vx3)*/+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNE+f_TSW))/(one+q) - c1over216 * drho;
      }

      q = q_dirTSW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k];
		 VeloY = vx2 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k];
		 un = fabs(-(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nx_dirTSW[k] 
				   -(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * ny_dirTSW[k] 
				   +(vx1 * nx_dirTSW[k] + vx2 * ny_dirTSW[k] + vx3 * nz_dirTSW[k]) * nz_dirTSW[k]);
		 ut = fabs(-VeloX - VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTSW[k] - ny_dirTSW[k] + nz_dirTSW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over216*(drho/*+three*(-vx1-vx2+vx3)*/+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSW+f_BNE))/(one+q) - c1over216 * drho;
      }

      q = q_dirTSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k];
		 VeloY = vx2 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k];
		 un = fabs(+(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nx_dirTSE[k] 
				   -(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * ny_dirTSE[k] 
				   +(vx1 * nx_dirTSE[k] + vx2 * ny_dirTSE[k] + vx3 * nz_dirTSE[k]) * nz_dirTSE[k]);
		 ut = fabs(+VeloX - VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirTSE[k] - ny_dirTSE[k] + nz_dirTSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over216*(drho/*+three*( vx1-vx2+vx3)*/+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSE+f_BNW))/(one+q) - c1over216 * drho;
      }

      q = q_dirBNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k];
		 VeloY = vx2 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k];
		 un = fabs(-(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nx_dirBNW[k] 
				   +(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * ny_dirBNW[k] 
				   -(vx1 * nx_dirBNW[k] + vx2 * ny_dirBNW[k] + vx3 * nz_dirBNW[k]) * nz_dirBNW[k]);
		 ut = fabs(-VeloX + VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirBNW[k] + ny_dirBNW[k] - nz_dirBNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over216*(drho/*+three*(-vx1+vx2-vx3)*/+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNW+f_TSE))/(one+q) - c1over216 * drho;
      }

      q = q_dirBSE[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k];
		 VeloY = vx2 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k];
		 VeloZ = vx3 - (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k];
		 un = fabs( (vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nx_dirBSE[k] 
				   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * ny_dirBSE[k] 
				   -(vx1 * nx_dirBSE[k] + vx2 * ny_dirBSE[k] + vx3 * nz_dirBSE[k]) * nz_dirBSE[k]);
		 ut = fabs( VeloX - VeloY - VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs( nx_dirBSE[k] - ny_dirBSE[k] - nz_dirBSE[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over216*(drho/*+three*( vx1-vx2-vx3)*/+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSE+f_TNW))/(one+q) - c1over216 * drho;
      }

      q = q_dirTNW[k];
      if (q>=zero && q<=one)
      {
		 VeloX = vx1 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k];
		 VeloY = vx2 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k];
		 VeloZ = vx3 - (vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k];
		 un = fabs(-(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nx_dirTNW[k] 
				   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * ny_dirTNW[k] 
				   +(vx1 * nx_dirTNW[k] + vx2 * ny_dirTNW[k] + vx3 * nz_dirTNW[k]) * nz_dirTNW[k]);
		 ut = fabs(-VeloX + VeloY + VeloZ);
		 tangential = ut / (ut + un + smallSingle);
		 qSlip = sliplength * fabs(-nx_dirTNW[k] + ny_dirTNW[k] + nz_dirTNW[k]);		//sliplength * e_i * n_i
		 //qSlip = (qSlip < zero) ? zero:qSlip;
		 //tangential = (tangential > one) ? one:tangential;
		 q = (q + qSlip)/(one + qSlip * (one - tangential) / (smallSingle + q));
         feq=c1over216*(drho/*+three*(-vx1+vx2+vx3)*/+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNW+f_BSE))/(one+q) - c1over216 * drho;
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


