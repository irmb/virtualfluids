#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/D3Q27.h"
#include "math.h"
#include "GPU/constant.h"

////////////////////////////////////////////////////////////////////////////////
__global__ void LB_Kernel_BGK_SP_Simple_27(doubflo omega,
                                                      unsigned int* bcMatD,
                                                      unsigned int* neighborX,
                                                      unsigned int* neighborY,
                                                      unsigned int* neighborZ,
                                                      doubflo* DDStart,
                                                      int size_Mat,
                                                      bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   bcMatD[k];

      if( (BC != GEO_SOLID) && (BC != GEO_VOID))
      {
         Distributions27 D;
         if (EvenOrOdd==true)
         {
            D.f[dirE   ] = &DDStart[dirE   *size_Mat];
            D.f[dirW   ] = &DDStart[dirW   *size_Mat];
            D.f[dirN   ] = &DDStart[dirN   *size_Mat];
            D.f[dirS   ] = &DDStart[dirS   *size_Mat];
            D.f[dirT   ] = &DDStart[dirT   *size_Mat];
            D.f[dirB   ] = &DDStart[dirB   *size_Mat];
            D.f[dirNE  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirSW  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirTNE ] = &DDStart[dirTNE *size_Mat];
            D.f[dirTSW ] = &DDStart[dirTSW *size_Mat];
            D.f[dirTSE ] = &DDStart[dirTSE *size_Mat];
            D.f[dirTNW ] = &DDStart[dirTNW *size_Mat];
            D.f[dirBNE ] = &DDStart[dirBNE *size_Mat];
            D.f[dirBSW ] = &DDStart[dirBSW *size_Mat];
            D.f[dirBSE ] = &DDStart[dirBSE *size_Mat];
            D.f[dirBNW ] = &DDStart[dirBNW *size_Mat];
         }
         else
         {
            D.f[dirW   ] = &DDStart[dirE   *size_Mat];
            D.f[dirE   ] = &DDStart[dirW   *size_Mat];
            D.f[dirS   ] = &DDStart[dirN   *size_Mat];
            D.f[dirN   ] = &DDStart[dirS   *size_Mat];
            D.f[dirB   ] = &DDStart[dirT   *size_Mat];
            D.f[dirT   ] = &DDStart[dirB   *size_Mat];
            D.f[dirSW  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirNE  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirBSW ] = &DDStart[dirTNE *size_Mat];
            D.f[dirBNE ] = &DDStart[dirTSW *size_Mat];
            D.f[dirBNW ] = &DDStart[dirTSE *size_Mat];
            D.f[dirBSE ] = &DDStart[dirTNW *size_Mat];
            D.f[dirTSW ] = &DDStart[dirBNE *size_Mat];
            D.f[dirTNE ] = &DDStart[dirBSW *size_Mat];
            D.f[dirTNW ] = &DDStart[dirBSE *size_Mat];
            D.f[dirTSE ] = &DDStart[dirBNW *size_Mat];
         }

         ////////////////////////////////////////////////////////////////////////////////
         //index
         //unsigned int kzero= k;
         //unsigned int ke   = k;
         unsigned int kw   = neighborX[k];
         //unsigned int kn   = k;
         unsigned int ks   = neighborY[k];
         //unsigned int kt   = k;
         unsigned int kb   = neighborZ[k];
         unsigned int ksw  = neighborY[kw];
         //unsigned int kne  = k;
         //unsigned int kse  = ks;
         //unsigned int knw  = kw;
         unsigned int kbw  = neighborZ[kw];
         //unsigned int kte  = k;
         //unsigned int kbe  = kb;
         //unsigned int ktw  = kw;
         unsigned int kbs  = neighborZ[ks];
         //unsigned int ktn  = k;
         //unsigned int kbn  = kb;
         //unsigned int kts  = ks;
         //unsigned int ktse = ks;
         //unsigned int kbnw = kbw;
         //unsigned int ktnw = kw;
         //unsigned int kbse = kbs;
         //unsigned int ktsw = ksw;
         //unsigned int kbne = kb;
         //unsigned int ktne = k;
         unsigned int kbsw = neighborZ[ksw];
         //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         doubflo fE    =  (D.f[dirE   ])[k  ];//ke
         doubflo fW    =  (D.f[dirW   ])[kw ];
         doubflo fN    =  (D.f[dirN   ])[k  ];//kn
         doubflo fS    =  (D.f[dirS   ])[ks ];
         doubflo fT    =  (D.f[dirT   ])[k  ];//kt
         doubflo fB    =  (D.f[dirB   ])[kb ];
         doubflo fNE   =  (D.f[dirNE  ])[k  ];//kne
         doubflo fSW   =  (D.f[dirSW  ])[ksw];
         doubflo fSE   =  (D.f[dirSE  ])[ks ];//kse
         doubflo fNW   =  (D.f[dirNW  ])[kw ];//knw
         doubflo fTE   =  (D.f[dirTE  ])[k  ];//kte
         doubflo fBW   =  (D.f[dirBW  ])[kbw];
         doubflo fBE   =  (D.f[dirBE  ])[kb ];//kbe
         doubflo fTW   =  (D.f[dirTW  ])[kw ];//ktw
         doubflo fTN   =  (D.f[dirTN  ])[k  ];//ktn
         doubflo fBS   =  (D.f[dirBS  ])[kbs];
         doubflo fBN   =  (D.f[dirBN  ])[kb ];//kbn
         doubflo fTS   =  (D.f[dirTS  ])[ks ];//kts
         doubflo fZERO =  (D.f[dirZERO])[k  ];//kzero
         doubflo fTNE   = (D.f[dirTNE ])[k  ];//ktne
         doubflo fTSW   = (D.f[dirTSW ])[ksw];//ktsw
         doubflo fTSE   = (D.f[dirTSE ])[ks ];//ktse
         doubflo fTNW   = (D.f[dirTNW ])[kw ];//ktnw
         doubflo fBNE   = (D.f[dirBNE ])[kb ];//kbne
         doubflo fBSW   = (D.f[dirBSW ])[kbsw];
         doubflo fBSE   = (D.f[dirBSE ])[kbs];//kbse
         doubflo fBNW   = (D.f[dirBNW ])[kbw];//kbnw
         ////////////////////////////////////////////////////////////////////////////////



         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         doubflo drho    =  (fTNE+fBSW)+(fTSW+fBNE)+(fTSE+fBNW)+(fTNW+fBSE)+(fNE+fSW)+(fNW+fSE)+(fTE+fBW)+(fBE+fTW)+(fTN+fBS)+(fBN+fTS)+(fE+fW)+(fN+fS)+(fT+fB)+fZERO;
         doubflo rho     =  drho + one;
         doubflo OORho   =  one/rho;
         doubflo vx1     =  OORho*((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
         doubflo vx2     =  OORho*((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
         doubflo vx3     =  OORho*((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
         doubflo cusq    =  c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

         fZERO = fZERO *(one+(-omega))-(-omega)*    c8over27* (drho-cusq);
         fE    = fE    *(one+(-omega))-(-omega)*    c2over27* (drho+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cusq);
         fW    = fW    *(one+(-omega))-(-omega)*    c2over27* (drho+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cusq);
         fN    = fN    *(one+(-omega))-(-omega)*    c2over27* (drho+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cusq);
         fS    = fS    *(one+(-omega))-(-omega)*    c2over27* (drho+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cusq);
         fT    = fT    *(one+(-omega))-(-omega)*    c2over27* (drho+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cusq);
         fB    = fB    *(one+(-omega))-(-omega)*    c2over27* (drho+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cusq);
         fNE   = fNE   *(one+(-omega))-(-omega)*    c1over54* (drho+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cusq);
         fSW   = fSW   *(one+(-omega))-(-omega)*    c1over54* (drho+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cusq);
         fSE   = fSE   *(one+(-omega))-(-omega)*    c1over54* (drho+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cusq);
         fNW   = fNW   *(one+(-omega))-(-omega)*    c1over54* (drho+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cusq);
         fTE   = fTE   *(one+(-omega))-(-omega)*    c1over54* (drho+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cusq);
         fBW   = fBW   *(one+(-omega))-(-omega)*    c1over54* (drho+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cusq);
         fBE   = fBE   *(one+(-omega))-(-omega)*    c1over54* (drho+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cusq);
         fTW   = fTW   *(one+(-omega))-(-omega)*    c1over54* (drho+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cusq);
         fTN   = fTN   *(one+(-omega))-(-omega)*    c1over54* (drho+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cusq);
         fBS   = fBS   *(one+(-omega))-(-omega)*    c1over54* (drho+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cusq);
         fBN   = fBN   *(one+(-omega))-(-omega)*    c1over54* (drho+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cusq);
         fTS   = fTS   *(one+(-omega))-(-omega)*    c1over54* (drho+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cusq);
         fTNE  = fTNE  *(one+(-omega))-(-omega)*    c1over216*(drho+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cusq);
         fBSW  = fBSW  *(one+(-omega))-(-omega)*    c1over216*(drho+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cusq);
         fBNE  = fBNE  *(one+(-omega))-(-omega)*    c1over216*(drho+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cusq);
         fTSW  = fTSW  *(one+(-omega))-(-omega)*    c1over216*(drho+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cusq);
         fTSE  = fTSE  *(one+(-omega))-(-omega)*    c1over216*(drho+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cusq);
         fBNW  = fBNW  *(one+(-omega))-(-omega)*    c1over216*(drho+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cusq);
         fBSE  = fBSE  *(one+(-omega))-(-omega)*    c1over216*(drho+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cusq);
         fTNW  = fTNW  *(one+(-omega))-(-omega)*    c1over216*(drho+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cusq);
         //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         (D.f[ dirE   ])[k   ] = fW    ;                                                                     
         (D.f[ dirW   ])[kw  ] = fE    ;                                                                     
         (D.f[ dirN   ])[k   ] = fS    ;
         (D.f[ dirS   ])[ks  ] = fN    ;
         (D.f[ dirT   ])[k   ] = fB    ;
         (D.f[ dirB   ])[kb  ] = fT    ;
         (D.f[ dirNE  ])[k   ] = fSW   ;
         (D.f[ dirSW  ])[ksw ] = fNE   ;
         (D.f[ dirSE  ])[ks  ] = fNW   ;
         (D.f[ dirNW  ])[kw  ] = fSE   ;
         (D.f[ dirTE  ])[k   ] = fBW   ;
         (D.f[ dirBW  ])[kbw ] = fTE   ;
         (D.f[ dirBE  ])[kb  ] = fTW   ;
         (D.f[ dirTW  ])[kw  ] = fBE   ;
         (D.f[ dirTN  ])[k   ] = fBS   ;
         (D.f[ dirBS  ])[kbs ] = fTN   ;
         (D.f[ dirBN  ])[kb  ] = fTS   ;
         (D.f[ dirTS  ])[ks  ] = fBN   ;
         (D.f[ dirZERO])[k   ] = fZERO ;
         (D.f[ dirTNE ])[k   ] = fBSW  ;
         (D.f[ dirTSE ])[ks  ] = fBNW  ;
         (D.f[ dirBNE ])[kb  ] = fTSW  ;
         (D.f[ dirBSE ])[kbs ] = fTNW  ;
         (D.f[ dirTNW ])[kw  ] = fBSE  ;
         (D.f[ dirTSW ])[ksw ] = fBNE  ;
         (D.f[ dirBNW ])[kbw ] = fTSE  ;
         (D.f[ dirBSW ])[kbsw] = fTNE  ;
      }                                                                                                                    
   }
}
////////////////////////////////////////////////////////////////////////////////


