#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/D3Q27.h"
//#include "LBM/LB.h"
#include "GPU/constant.h"

////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMac27( real* vxD,
                                        real* vyD,
                                        real* vzD,
                                        real* rhoD,
                                        unsigned int* geoD,
                                        unsigned int* neighborX,
                                        unsigned int* neighborY,
                                        unsigned int* neighborZ,
                                        unsigned int size_Mat,
                                        real* DD,
                                        bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   unsigned int  k;                   // Zugriff auf arrays im device
   //
   unsigned int tx = threadIdx.x;     // Thread index = lokaler i index
   unsigned int by = blockIdx.x;      // Block index x
   unsigned int bz = blockIdx.y;      // Block index y
   unsigned int  x = tx + STARTOFFX;  // Globaler x-Index 
   unsigned int  y = by + STARTOFFY;  // Globaler y-Index 
   unsigned int  z = bz + STARTOFFZ;  // Globaler z-Index 

   const unsigned sizeX = blockDim.x;
   const unsigned sizeY = gridDim.x;
   const unsigned nx = sizeX + 2 * STARTOFFX;
   const unsigned ny = sizeY + 2 * STARTOFFY;

   k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////
   //index
   unsigned int kzero= k;
   unsigned int ke   = k;
   unsigned int kw   = neighborX[k];
   unsigned int kn   = k;
   unsigned int ks   = neighborY[k];
   unsigned int kt   = k;
   unsigned int kb   = neighborZ[k];
   unsigned int ksw  = neighborY[kw];
   unsigned int kne  = k;
   unsigned int kse  = ks;
   unsigned int knw  = kw;
   unsigned int kbw  = neighborZ[kw];
   unsigned int kte  = k;
   unsigned int kbe  = kb;
   unsigned int ktw  = kw;
   unsigned int kbs  = neighborZ[ks];
   unsigned int ktn  = k;
   unsigned int kbn  = kb;
   unsigned int kts  = ks;
   unsigned int ktse = ks;
   unsigned int kbnw = kbw;
   unsigned int ktnw = kw;
   unsigned int kbse = kbs;
   unsigned int ktsw = ksw;
   unsigned int kbne = kb;
   unsigned int ktne = k;
   unsigned int kbsw = neighborZ[ksw];
   //unsigned int nxny = nx*ny;
   //unsigned int kzero= k;
   //unsigned int ke   = k;
   //unsigned int kw   = k + 1;
   //unsigned int kn   = k;
   //unsigned int ks   = k + nx;
   //unsigned int kt   = k;
   //unsigned int kb   = k + nxny;
   //unsigned int ksw  = k + nx + 1;
   //unsigned int kne  = k;
   //unsigned int kse  = k + nx;
   //unsigned int knw  = k + 1;
   //unsigned int kbw  = k + nxny + 1;
   //unsigned int kte  = k;
   //unsigned int kbe  = k + nxny;
   //unsigned int ktw  = k + 1;
   //unsigned int kbs  = k + nxny + nx;
   //unsigned int ktn  = k;
   //unsigned int kbn  = k + nxny;
   //unsigned int kts  = k + nx;
   //unsigned int ktse = k + nx;
   //unsigned int kbnw = k + nxny + 1;
   //unsigned int ktnw = k + 1;
   //unsigned int kbse = k + nxny + nx;
   //unsigned int ktsw = k + nx + 1;
   //unsigned int kbne = k + nxny;
   //unsigned int ktne = k;
   //unsigned int kbsw = k + nxny + nx + 1;
   //////////////////////////////////////////////////////////////////////////
   rhoD[k] = zero;
   vxD[k]  = zero;
   vyD[k]  = zero;
   vzD[k]  = zero;

   if(geoD[k] == GEO_FLUID)
   {
      rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                     (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                     (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                     (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                     (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                     (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                     (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                     (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                     (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                     (D.f[dirZERO])[kzero]+ 
                     (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                     (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                     (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                     (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

      vxD[k]     =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                     (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                     (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                     (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                     (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                     (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                     (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                     (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                     (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

      vyD[k]     =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                     (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                     (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                     (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                     (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                     (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                     (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                     (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                     (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

      vzD[k]     =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                     (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                     (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                     (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                     (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                     (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                     (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                     (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                     (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];
   }
}





////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacSP27( real* vxD,
                                          real* vyD,
                                          real* vzD,
                                          real* rhoD,
                                          real* pressD,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD,
                                          bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = zero;
	  rhoD[k]   = zero;
	  vxD[k]    = zero;
	  vyD[k]    = zero;
	  vzD[k]    = zero;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vxD[k]     =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         vyD[k]     =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vzD[k]     =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         pressD[k]  =  ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        2.f*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        3.f*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (one+zero*rhoD[k])) * c1o2+rhoD[k]; // times zero for incompressible case   
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5

      }
   }
}




























////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacCompSP27( real* vxD,
											  real* vyD,
											  real* vzD,
											  real* rhoD,
											  real* pressD,
											  unsigned int* geoD,
											  unsigned int* neighborX,
											  unsigned int* neighborY,
											  unsigned int* neighborZ,
											  unsigned int size_Mat,
											  real* DD,
											  bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = zero;
	  rhoD[k]   = zero;
	  vxD[k]    = zero;
	  vyD[k]    = zero;
	  vzD[k]    = zero;

      if(geoD[k] == GEO_FLUID || geoD[k] == GEO_PM_0)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vxD[k]     =  ((D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
						(D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (one + rhoD[k]);

         vyD[k]     =  ((D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]) / (one + rhoD[k]);

         vzD[k]     =  ((D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (one + rhoD[k]);

         pressD[k]  =  ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        two*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        three*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (one+rhoD[k])) * c1o2+rhoD[k]; // times zero for incompressible case   
         //achtung op hart gesetzt Annahme op = 1 ;                                                      ^^^^(1.0/op-0.5)=0.5

      }
   }
}



























////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacThS7( real* Conc,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD7,
                                          bool evenOrOdd)
{
   Distributions7 D7;
   if (evenOrOdd==true)
   {
      D7.f[0] = &DD7[0*size_Mat];
      D7.f[1] = &DD7[1*size_Mat];
      D7.f[2] = &DD7[2*size_Mat];
      D7.f[3] = &DD7[3*size_Mat];
      D7.f[4] = &DD7[4*size_Mat];
      D7.f[5] = &DD7[5*size_Mat];
      D7.f[6] = &DD7[6*size_Mat];
   } 
   else
   {
      D7.f[0] = &DD7[0*size_Mat];
      D7.f[2] = &DD7[1*size_Mat];
      D7.f[1] = &DD7[2*size_Mat];
      D7.f[4] = &DD7[3*size_Mat];
      D7.f[3] = &DD7[4*size_Mat];
      D7.f[6] = &DD7[5*size_Mat];
      D7.f[5] = &DD7[6*size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = zero;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D7.f[1])[ke   ]+ (D7.f[2])[kw  ]+ 
                        (D7.f[3])[kn   ]+ (D7.f[4])[ks  ]+
                        (D7.f[5])[kt   ]+ (D7.f[6])[kb  ]+
                        (D7.f[0])[kzero];  
      }
   }
}





























////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void GetPlaneConcThS7(real* Conc,
								            int* kPC,
								            unsigned int numberOfPointskPC,
											unsigned int* geoD,
											unsigned int* neighborX,
											unsigned int* neighborY,
											unsigned int* neighborZ,
											unsigned int size_Mat,
											real* DD7,
											bool evenOrOdd)
{
   Distributions7 D7;
   if (evenOrOdd==true)
   {
      D7.f[0] = &DD7[0*size_Mat];
      D7.f[1] = &DD7[1*size_Mat];
      D7.f[2] = &DD7[2*size_Mat];
      D7.f[3] = &DD7[3*size_Mat];
      D7.f[4] = &DD7[4*size_Mat];
      D7.f[5] = &DD7[5*size_Mat];
      D7.f[6] = &DD7[6*size_Mat];
   } 
   else
   {
      D7.f[0] = &DD7[0*size_Mat];
      D7.f[2] = &DD7[1*size_Mat];
      D7.f[1] = &DD7[2*size_Mat];
      D7.f[4] = &DD7[3*size_Mat];
      D7.f[3] = &DD7[4*size_Mat];
      D7.f[6] = &DD7[5*size_Mat];
      D7.f[5] = &DD7[6*size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfPointskPC)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kPC[k];
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = zero;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D7.f[1])[ke   ]+ (D7.f[2])[kw  ]+ 
                        (D7.f[3])[kn   ]+ (D7.f[4])[ks  ]+
                        (D7.f[5])[kt   ]+ (D7.f[6])[kb  ]+
                        (D7.f[0])[kzero];  
      }
   }
}




































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void GetPlaneConcThS27(real* Conc,
								             int* kPC,
								             unsigned int numberOfPointskPC,
											 unsigned int* geoD,
											 unsigned int* neighborX,
											 unsigned int* neighborY,
											 unsigned int* neighborZ,
											 unsigned int size_Mat,
											 real* DD27,
											 bool evenOrOdd)
{
   Distributions27 D27;
   if (evenOrOdd==true)
   {
      D27.f[dirE   ] = &DD27[dirE   *size_Mat];
      D27.f[dirW   ] = &DD27[dirW   *size_Mat];
      D27.f[dirN   ] = &DD27[dirN   *size_Mat];
      D27.f[dirS   ] = &DD27[dirS   *size_Mat];
      D27.f[dirT   ] = &DD27[dirT   *size_Mat];
      D27.f[dirB   ] = &DD27[dirB   *size_Mat];
      D27.f[dirNE  ] = &DD27[dirNE  *size_Mat];
      D27.f[dirSW  ] = &DD27[dirSW  *size_Mat];
      D27.f[dirSE  ] = &DD27[dirSE  *size_Mat];
      D27.f[dirNW  ] = &DD27[dirNW  *size_Mat];
      D27.f[dirTE  ] = &DD27[dirTE  *size_Mat];
      D27.f[dirBW  ] = &DD27[dirBW  *size_Mat];
      D27.f[dirBE  ] = &DD27[dirBE  *size_Mat];
      D27.f[dirTW  ] = &DD27[dirTW  *size_Mat];
      D27.f[dirTN  ] = &DD27[dirTN  *size_Mat];
      D27.f[dirBS  ] = &DD27[dirBS  *size_Mat];
      D27.f[dirBN  ] = &DD27[dirBN  *size_Mat];
      D27.f[dirTS  ] = &DD27[dirTS  *size_Mat];
      D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
      D27.f[dirTNE ] = &DD27[dirTNE *size_Mat];
      D27.f[dirTSW ] = &DD27[dirTSW *size_Mat];
      D27.f[dirTSE ] = &DD27[dirTSE *size_Mat];
      D27.f[dirTNW ] = &DD27[dirTNW *size_Mat];
      D27.f[dirBNE ] = &DD27[dirBNE *size_Mat];
      D27.f[dirBSW ] = &DD27[dirBSW *size_Mat];
      D27.f[dirBSE ] = &DD27[dirBSE *size_Mat];
      D27.f[dirBNW ] = &DD27[dirBNW *size_Mat];
   }
   else
   {
      D27.f[dirW   ] = &DD27[dirE   *size_Mat];
      D27.f[dirE   ] = &DD27[dirW   *size_Mat];
      D27.f[dirS   ] = &DD27[dirN   *size_Mat];
      D27.f[dirN   ] = &DD27[dirS   *size_Mat];
      D27.f[dirB   ] = &DD27[dirT   *size_Mat];
      D27.f[dirT   ] = &DD27[dirB   *size_Mat];
      D27.f[dirSW  ] = &DD27[dirNE  *size_Mat];
      D27.f[dirNE  ] = &DD27[dirSW  *size_Mat];
      D27.f[dirNW  ] = &DD27[dirSE  *size_Mat];
      D27.f[dirSE  ] = &DD27[dirNW  *size_Mat];
      D27.f[dirBW  ] = &DD27[dirTE  *size_Mat];
      D27.f[dirTE  ] = &DD27[dirBW  *size_Mat];
      D27.f[dirTW  ] = &DD27[dirBE  *size_Mat];
      D27.f[dirBE  ] = &DD27[dirTW  *size_Mat];
      D27.f[dirBS  ] = &DD27[dirTN  *size_Mat];
      D27.f[dirTN  ] = &DD27[dirBS  *size_Mat];
      D27.f[dirTS  ] = &DD27[dirBN  *size_Mat];
      D27.f[dirBN  ] = &DD27[dirTS  *size_Mat];
      D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
      D27.f[dirBSW ] = &DD27[dirTNE *size_Mat];
      D27.f[dirBNE ] = &DD27[dirTSW *size_Mat];
      D27.f[dirBNW ] = &DD27[dirTSE *size_Mat];
      D27.f[dirBSE ] = &DD27[dirTNW *size_Mat];
      D27.f[dirTSW ] = &DD27[dirBNE *size_Mat];
      D27.f[dirTNE ] = &DD27[dirBSW *size_Mat];
      D27.f[dirTNW ] = &DD27[dirBSE *size_Mat];
      D27.f[dirTSE ] = &DD27[dirBNW *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfPointskPC)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kPC[k];
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = kzero;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = kzero;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = kzero;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = kzero;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = zero;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D27.f[dirE   ])[ke  ]+ (D27.f[dirW   ])[kw  ]+ 
                        (D27.f[dirN   ])[kn  ]+ (D27.f[dirS   ])[ks  ]+
                        (D27.f[dirT   ])[kt  ]+ (D27.f[dirB   ])[kb  ]+
                        (D27.f[dirNE  ])[kne ]+ (D27.f[dirSW  ])[ksw ]+
                        (D27.f[dirSE  ])[kse ]+ (D27.f[dirNW  ])[knw ]+
                        (D27.f[dirTE  ])[kte ]+ (D27.f[dirBW  ])[kbw ]+
                        (D27.f[dirBE  ])[kbe ]+ (D27.f[dirTW  ])[ktw ]+
                        (D27.f[dirTN  ])[ktn ]+ (D27.f[dirBS  ])[kbs ]+
                        (D27.f[dirBN  ])[kbn ]+ (D27.f[dirTS  ])[kts ]+
                        (D27.f[dirZERO])[kzero]+ 
                        (D27.f[dirTNE ])[ktne]+ (D27.f[dirTSW ])[ktsw]+
                        (D27.f[dirTSE ])[ktse]+ (D27.f[dirTNW ])[ktnw]+
                        (D27.f[dirBNE ])[kbne]+ (D27.f[dirBSW ])[kbsw]+
                        (D27.f[dirBSE ])[kbse]+ (D27.f[dirBNW ])[kbnw];
      }
   }   
}




































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacThS27(real* Conc,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD27,
                                          bool evenOrOdd)
{
   Distributions27 D27;
   if (evenOrOdd==true)
   {
      D27.f[dirE   ] = &DD27[dirE   *size_Mat];
      D27.f[dirW   ] = &DD27[dirW   *size_Mat];
      D27.f[dirN   ] = &DD27[dirN   *size_Mat];
      D27.f[dirS   ] = &DD27[dirS   *size_Mat];
      D27.f[dirT   ] = &DD27[dirT   *size_Mat];
      D27.f[dirB   ] = &DD27[dirB   *size_Mat];
      D27.f[dirNE  ] = &DD27[dirNE  *size_Mat];
      D27.f[dirSW  ] = &DD27[dirSW  *size_Mat];
      D27.f[dirSE  ] = &DD27[dirSE  *size_Mat];
      D27.f[dirNW  ] = &DD27[dirNW  *size_Mat];
      D27.f[dirTE  ] = &DD27[dirTE  *size_Mat];
      D27.f[dirBW  ] = &DD27[dirBW  *size_Mat];
      D27.f[dirBE  ] = &DD27[dirBE  *size_Mat];
      D27.f[dirTW  ] = &DD27[dirTW  *size_Mat];
      D27.f[dirTN  ] = &DD27[dirTN  *size_Mat];
      D27.f[dirBS  ] = &DD27[dirBS  *size_Mat];
      D27.f[dirBN  ] = &DD27[dirBN  *size_Mat];
      D27.f[dirTS  ] = &DD27[dirTS  *size_Mat];
      D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
      D27.f[dirTNE ] = &DD27[dirTNE *size_Mat];
      D27.f[dirTSW ] = &DD27[dirTSW *size_Mat];
      D27.f[dirTSE ] = &DD27[dirTSE *size_Mat];
      D27.f[dirTNW ] = &DD27[dirTNW *size_Mat];
      D27.f[dirBNE ] = &DD27[dirBNE *size_Mat];
      D27.f[dirBSW ] = &DD27[dirBSW *size_Mat];
      D27.f[dirBSE ] = &DD27[dirBSE *size_Mat];
      D27.f[dirBNW ] = &DD27[dirBNW *size_Mat];
   }
   else
   {
      D27.f[dirW   ] = &DD27[dirE   *size_Mat];
      D27.f[dirE   ] = &DD27[dirW   *size_Mat];
      D27.f[dirS   ] = &DD27[dirN   *size_Mat];
      D27.f[dirN   ] = &DD27[dirS   *size_Mat];
      D27.f[dirB   ] = &DD27[dirT   *size_Mat];
      D27.f[dirT   ] = &DD27[dirB   *size_Mat];
      D27.f[dirSW  ] = &DD27[dirNE  *size_Mat];
      D27.f[dirNE  ] = &DD27[dirSW  *size_Mat];
      D27.f[dirNW  ] = &DD27[dirSE  *size_Mat];
      D27.f[dirSE  ] = &DD27[dirNW  *size_Mat];
      D27.f[dirBW  ] = &DD27[dirTE  *size_Mat];
      D27.f[dirTE  ] = &DD27[dirBW  *size_Mat];
      D27.f[dirTW  ] = &DD27[dirBE  *size_Mat];
      D27.f[dirBE  ] = &DD27[dirTW  *size_Mat];
      D27.f[dirBS  ] = &DD27[dirTN  *size_Mat];
      D27.f[dirTN  ] = &DD27[dirBS  *size_Mat];
      D27.f[dirTS  ] = &DD27[dirBN  *size_Mat];
      D27.f[dirBN  ] = &DD27[dirTS  *size_Mat];
      D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
      D27.f[dirBSW ] = &DD27[dirTNE *size_Mat];
      D27.f[dirBNE ] = &DD27[dirTSW *size_Mat];
      D27.f[dirBNW ] = &DD27[dirTSE *size_Mat];
      D27.f[dirBSE ] = &DD27[dirTNW *size_Mat];
      D27.f[dirTSW ] = &DD27[dirBNE *size_Mat];
      D27.f[dirTNE ] = &DD27[dirBSW *size_Mat];
      D27.f[dirTNW ] = &DD27[dirBSE *size_Mat];
      D27.f[dirTSE ] = &DD27[dirBNW *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      Conc[k] = zero;

      if(geoD[k] == GEO_FLUID)
      {
         Conc[k]    =   (D27.f[dirE   ])[ke  ]+ (D27.f[dirW   ])[kw  ]+ 
                        (D27.f[dirN   ])[kn  ]+ (D27.f[dirS   ])[ks  ]+
                        (D27.f[dirT   ])[kt  ]+ (D27.f[dirB   ])[kb  ]+
                        (D27.f[dirNE  ])[kne ]+ (D27.f[dirSW  ])[ksw ]+
                        (D27.f[dirSE  ])[kse ]+ (D27.f[dirNW  ])[knw ]+
                        (D27.f[dirTE  ])[kte ]+ (D27.f[dirBW  ])[kbw ]+
                        (D27.f[dirBE  ])[kbe ]+ (D27.f[dirTW  ])[ktw ]+
                        (D27.f[dirTN  ])[ktn ]+ (D27.f[dirBS  ])[kbs ]+
                        (D27.f[dirBN  ])[kbn ]+ (D27.f[dirTS  ])[kts ]+
                        (D27.f[dirZERO])[kzero]+ 
                        (D27.f[dirTNE ])[ktne]+ (D27.f[dirTSW ])[ktsw]+
                        (D27.f[dirTSE ])[ktse]+ (D27.f[dirTNW ])[ktnw]+
                        (D27.f[dirBNE ])[kbne]+ (D27.f[dirBSW ])[kbsw]+
                        (D27.f[dirBSE ])[kbse]+ (D27.f[dirBNW ])[kbnw];
      }
   }   
}




















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMedSP27( real* vxD,
                                          real* vyD,
                                          real* vzD,
                                          real* rhoD,
                                          real* pressD,
                                          unsigned int* geoD,
                                          unsigned int* neighborX,
                                          unsigned int* neighborY,
                                          unsigned int* neighborZ,
                                          unsigned int size_Mat,
                                          real* DD,
                                          bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = zero;
	  rhoD[k]   = zero;
	  vxD[k]    = zero;
	  vyD[k]    = zero;
	  vzD[k]    = zero;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]+
                        RHO;

         vxD[k]     =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]+
                        VX;

         vyD[k]     =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]+
                        VY;

         vzD[k]     =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]+
                        VZ;

         pressD[k]  =   ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        two*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        three*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (one+rhoD[k])) * c1o2+rhoD[k]+
                        PRESS;    
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMedCompSP27( real* vxD,
											  real* vyD,
											  real* vzD,
											  real* rhoD,
											  real* pressD,
											  unsigned int* geoD,
											  unsigned int* neighborX,
											  unsigned int* neighborY,
											  unsigned int* neighborZ,
											  unsigned int size_Mat,
											  real* DD,
											  bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= k;
      unsigned int ke   = k;
      unsigned int kw   = neighborX[k];
      unsigned int kn   = k;
      unsigned int ks   = neighborY[k];
      unsigned int kt   = k;
      unsigned int kb   = neighborZ[k];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = k;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = k;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = k;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = k;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = zero;
	  rhoD[k]   = zero;
	  vxD[k]    = zero;
	  vyD[k]    = zero;
	  vzD[k]    = zero;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]+
                        RHO;

         vxD[k]     =  ((D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
                        VX;

         vyD[k]     =  ((D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
                        VY;

         vzD[k]     =  ((D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw]) / (one + rhoD[k])+
                        VZ;

         pressD[k]  =  ((D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        two*(
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ])+
                        three*(
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw])-
                        rhoD[k]-(vxD[k] * vxD[k] + vyD[k] * vyD[k] + vzD[k] * vzD[k]) * (one+rhoD[k])) * c1o2+rhoD[k]+
                        PRESS;    
         //achtung op hart gesetzt Annahme op = 1 ;                                                    ^^^^(1.0/op-0.5)=0.5
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMacMedSP27( real* vxD,
                                             real* vyD,
                                             real* vzD,
                                             real* rhoD,
                                             real* pressD,
                                             unsigned int* geoD,
                                             unsigned int* neighborX,
                                             unsigned int* neighborY,
                                             unsigned int* neighborZ,
                                             unsigned int tdiff,
                                             unsigned int size_Mat,
                                             bool evenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      //////////////////////////////////////////////////////////////////////////
      real PRESS = pressD[k];
      real RHO   = rhoD[k];
      real VX    = vxD[k];
      real VY    = vyD[k];
      real VZ    = vzD[k];
      //////////////////////////////////////////////////////////////////////////
      pressD[k] = zero;
      rhoD[k]   = zero;
      vxD[k]    = zero;
      vyD[k]    = zero;
      vzD[k]    = zero;

      if(geoD[k] == GEO_FLUID)
      {
         rhoD[k]    =   RHO   / tdiff;
         vxD[k]     =   VX    / tdiff;
         vyD[k]     =   VY    / tdiff;
         vzD[k]     =   VZ    / tdiff;
         pressD[k]  =   PRESS / tdiff;    
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





















////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBCalcMeasurePoints( real* vxMP,
												real* vyMP,
												real* vzMP,
												real* rhoMP,
												unsigned int* kMP,
												unsigned int numberOfPointskMP,
												unsigned int MPClockCycle,
												unsigned int t,
												unsigned int* geoD,
												unsigned int* neighborX,
												unsigned int* neighborY,
												unsigned int* neighborZ,
												unsigned int size_Mat,
												real* DD,
												bool evenOrOdd)
{
	Distributions27 D;
	if (evenOrOdd==true)
	{
		D.f[dirE   ] = &DD[dirE   *size_Mat];
		D.f[dirW   ] = &DD[dirW   *size_Mat];
		D.f[dirN   ] = &DD[dirN   *size_Mat];
		D.f[dirS   ] = &DD[dirS   *size_Mat];
		D.f[dirT   ] = &DD[dirT   *size_Mat];
		D.f[dirB   ] = &DD[dirB   *size_Mat];
		D.f[dirNE  ] = &DD[dirNE  *size_Mat];
		D.f[dirSW  ] = &DD[dirSW  *size_Mat];
		D.f[dirSE  ] = &DD[dirSE  *size_Mat];
		D.f[dirNW  ] = &DD[dirNW  *size_Mat];
		D.f[dirTE  ] = &DD[dirTE  *size_Mat];
		D.f[dirBW  ] = &DD[dirBW  *size_Mat];
		D.f[dirBE  ] = &DD[dirBE  *size_Mat];
		D.f[dirTW  ] = &DD[dirTW  *size_Mat];
		D.f[dirTN  ] = &DD[dirTN  *size_Mat];
		D.f[dirBS  ] = &DD[dirBS  *size_Mat];
		D.f[dirBN  ] = &DD[dirBN  *size_Mat];
		D.f[dirTS  ] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE ] = &DD[dirTNE *size_Mat];
		D.f[dirTSW ] = &DD[dirTSW *size_Mat];
		D.f[dirTSE ] = &DD[dirTSE *size_Mat];
		D.f[dirTNW ] = &DD[dirTNW *size_Mat];
		D.f[dirBNE ] = &DD[dirBNE *size_Mat];
		D.f[dirBSW ] = &DD[dirBSW *size_Mat];
		D.f[dirBSE ] = &DD[dirBSE *size_Mat];
		D.f[dirBNW ] = &DD[dirBNW *size_Mat];
	} 
	else
	{
		D.f[dirW   ] = &DD[dirE   *size_Mat];
		D.f[dirE   ] = &DD[dirW   *size_Mat];
		D.f[dirS   ] = &DD[dirN   *size_Mat];
		D.f[dirN   ] = &DD[dirS   *size_Mat];
		D.f[dirB   ] = &DD[dirT   *size_Mat];
		D.f[dirT   ] = &DD[dirB   *size_Mat];
		D.f[dirSW  ] = &DD[dirNE  *size_Mat];
		D.f[dirNE  ] = &DD[dirSW  *size_Mat];
		D.f[dirNW  ] = &DD[dirSE  *size_Mat];
		D.f[dirSE  ] = &DD[dirNW  *size_Mat];
		D.f[dirBW  ] = &DD[dirTE  *size_Mat];
		D.f[dirTE  ] = &DD[dirBW  *size_Mat];
		D.f[dirTW  ] = &DD[dirBE  *size_Mat];
		D.f[dirBE  ] = &DD[dirTW  *size_Mat];
		D.f[dirBS  ] = &DD[dirTN  *size_Mat];
		D.f[dirTN  ] = &DD[dirBS  *size_Mat];
		D.f[dirTS  ] = &DD[dirBN  *size_Mat];
		D.f[dirBN  ] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE ] = &DD[dirBSW *size_Mat];
		D.f[dirTSW ] = &DD[dirBNE *size_Mat];
		D.f[dirTSE ] = &DD[dirBNW *size_Mat];
		D.f[dirTNW ] = &DD[dirBSE *size_Mat];
		D.f[dirBNE ] = &DD[dirTSW *size_Mat];
		D.f[dirBSW ] = &DD[dirTNE *size_Mat];
		D.f[dirBSE ] = &DD[dirTNW *size_Mat];
		D.f[dirBNW ] = &DD[dirTSE *size_Mat];
	}
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if(k<numberOfPointskMP)
	{
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int kzero= kMP[k];//k;
      unsigned int ke   = kzero;
      unsigned int kw   = neighborX[kzero];
      unsigned int kn   = kzero;
      unsigned int ks   = neighborY[kzero];
      unsigned int kt   = kzero;
      unsigned int kb   = neighborZ[kzero];
      unsigned int ksw  = neighborY[kw];
      unsigned int kne  = kzero;
      unsigned int kse  = ks;
      unsigned int knw  = kw;
      unsigned int kbw  = neighborZ[kw];
      unsigned int kte  = kzero;
      unsigned int kbe  = kb;
      unsigned int ktw  = kw;
      unsigned int kbs  = neighborZ[ks];
      unsigned int ktn  = kzero;
      unsigned int kbn  = kb;
      unsigned int kts  = ks;
      unsigned int ktse = ks;
      unsigned int kbnw = kbw;
      unsigned int ktnw = kw;
      unsigned int kbse = kbs;
      unsigned int ktsw = ksw;
      unsigned int kbne = kb;
      unsigned int ktne = kzero;
      unsigned int kbsw = neighborZ[ksw];
      //////////////////////////////////////////////////////////////////////////
	  unsigned int kMac = k*MPClockCycle + t;
	  //////////////////////////////////////////////////////////////////////////

      if(geoD[kzero] == GEO_FLUID)
      {
         rhoMP[kMac]=   (D.f[dirE   ])[ke  ]+ (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirN   ])[kn  ]+ (D.f[dirS   ])[ks  ]+
                        (D.f[dirT   ])[kt  ]+ (D.f[dirB   ])[kb  ]+
                        (D.f[dirNE  ])[kne ]+ (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]+ (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]+ (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirZERO])[kzero]+ 
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]+ (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vxMP[kMac] =   (D.f[dirE   ])[ke  ]- (D.f[dirW   ])[kw  ]+ 
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]+
                        (D.f[dirSE  ])[kse ]- (D.f[dirNW  ])[knw ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]+
                        (D.f[dirBE  ])[kbe ]- (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]- (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]+ 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];

         vyMP[kMac] =   (D.f[dirN   ])[kn  ]- (D.f[dirS   ])[ks  ]+
                        (D.f[dirNE  ])[kne ]- (D.f[dirSW  ])[ksw ]-
                        (D.f[dirSE  ])[kse ]+ (D.f[dirNW  ])[knw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]+
                        (D.f[dirBN  ])[kbn ]- (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]- (D.f[dirTSW ])[ktsw]- 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]+ 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]+ (D.f[dirBNW ])[kbnw];

         vzMP[kMac] =   (D.f[dirT   ])[kt  ]- (D.f[dirB   ])[kb  ]+
                        (D.f[dirTE  ])[kte ]- (D.f[dirBW  ])[kbw ]-
                        (D.f[dirBE  ])[kbe ]+ (D.f[dirTW  ])[ktw ]+
                        (D.f[dirTN  ])[ktn ]- (D.f[dirBS  ])[kbs ]-
                        (D.f[dirBN  ])[kbn ]+ (D.f[dirTS  ])[kts ]+
                        (D.f[dirTNE ])[ktne]+ (D.f[dirTSW ])[ktsw]+ 
                        (D.f[dirTSE ])[ktse]+ (D.f[dirTNW ])[ktnw]- 
                        (D.f[dirBNE ])[kbne]- (D.f[dirBSW ])[kbsw]- 
                        (D.f[dirBSE ])[kbse]- (D.f[dirBNW ])[kbnw];
      }
   }
}
////////////////////////////////////////////////////////////////////////////////





































////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LBSetOutputWallVelocitySP27( real* vxD,
														real* vyD,
														real* vzD,
														real* vxWall,
														real* vyWall,
														real* vzWall,
														int numberOfWallNodes, 
														int* kWallNodes, 
														real* rhoD,
														real* pressD,
														unsigned int* geoD,
														unsigned int* neighborX,
														unsigned int* neighborY,
														unsigned int* neighborZ,
														unsigned int size_Mat,
														real* DD,
														bool evenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<numberOfWallNodes)
   {
      //////////////////////////////////////////////////////////////////////////
      //index
      unsigned int KWN  = kWallNodes[k];
      //////////////////////////////////////////////////////////////////////////
      vxD[KWN] = 0.0;//vxWall[k];
      vyD[KWN] = 0.0;//vyWall[k];
      vzD[KWN] = 0.0;//vzWall[k];
   }
}





























