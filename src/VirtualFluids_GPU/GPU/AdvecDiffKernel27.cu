#include "hip/hip_runtime.h"
/* Device code */
#include "LBM/D3Q27.h"
#include "math.h"
#include "GPU/constant.h"

////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LB_Kernel_ThS27(  doubflo diffusivity,
                                             unsigned int* bcMatD,
                                             unsigned int* neighborX,
                                             unsigned int* neighborY,
                                             unsigned int* neighborZ,
                                             doubflo* DDStart,
                                             doubflo* DD27,
                                             int size_Mat,
                                             bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   bcMatD[k];

      if( (BC != GEO_SOLID) && (BC != GEO_VOID))
      {
         Distributions27 D;
         if (EvenOrOdd==true)
         {
            D.f[dirE   ] = &DDStart[dirE   *size_Mat];
            D.f[dirW   ] = &DDStart[dirW   *size_Mat];
            D.f[dirN   ] = &DDStart[dirN   *size_Mat];
            D.f[dirS   ] = &DDStart[dirS   *size_Mat];
            D.f[dirT   ] = &DDStart[dirT   *size_Mat];
            D.f[dirB   ] = &DDStart[dirB   *size_Mat];
            D.f[dirNE  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirSW  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirTNE ] = &DDStart[dirTNE *size_Mat];
            D.f[dirTSW ] = &DDStart[dirTSW *size_Mat];
            D.f[dirTSE ] = &DDStart[dirTSE *size_Mat];
            D.f[dirTNW ] = &DDStart[dirTNW *size_Mat];
            D.f[dirBNE ] = &DDStart[dirBNE *size_Mat];
            D.f[dirBSW ] = &DDStart[dirBSW *size_Mat];
            D.f[dirBSE ] = &DDStart[dirBSE *size_Mat];
            D.f[dirBNW ] = &DDStart[dirBNW *size_Mat];
         }
         else
         {
            D.f[dirW   ] = &DDStart[dirE   *size_Mat];
            D.f[dirE   ] = &DDStart[dirW   *size_Mat];
            D.f[dirS   ] = &DDStart[dirN   *size_Mat];
            D.f[dirN   ] = &DDStart[dirS   *size_Mat];
            D.f[dirB   ] = &DDStart[dirT   *size_Mat];
            D.f[dirT   ] = &DDStart[dirB   *size_Mat];
            D.f[dirSW  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirNE  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirBSW ] = &DDStart[dirTNE *size_Mat];
            D.f[dirBNE ] = &DDStart[dirTSW *size_Mat];
            D.f[dirBNW ] = &DDStart[dirTSE *size_Mat];
            D.f[dirBSE ] = &DDStart[dirTNW *size_Mat];
            D.f[dirTSW ] = &DDStart[dirBNE *size_Mat];
            D.f[dirTNE ] = &DDStart[dirBSW *size_Mat];
            D.f[dirTNW ] = &DDStart[dirBSE *size_Mat];
            D.f[dirTSE ] = &DDStart[dirBNW *size_Mat];
         }

         Distributions27 D27;
         if (EvenOrOdd==true)
         {
            D27.f[dirE   ] = &DD27[dirE   *size_Mat];
            D27.f[dirW   ] = &DD27[dirW   *size_Mat];
            D27.f[dirN   ] = &DD27[dirN   *size_Mat];
            D27.f[dirS   ] = &DD27[dirS   *size_Mat];
            D27.f[dirT   ] = &DD27[dirT   *size_Mat];
            D27.f[dirB   ] = &DD27[dirB   *size_Mat];
            D27.f[dirNE  ] = &DD27[dirNE  *size_Mat];
            D27.f[dirSW  ] = &DD27[dirSW  *size_Mat];
            D27.f[dirSE  ] = &DD27[dirSE  *size_Mat];
            D27.f[dirNW  ] = &DD27[dirNW  *size_Mat];
            D27.f[dirTE  ] = &DD27[dirTE  *size_Mat];
            D27.f[dirBW  ] = &DD27[dirBW  *size_Mat];
            D27.f[dirBE  ] = &DD27[dirBE  *size_Mat];
            D27.f[dirTW  ] = &DD27[dirTW  *size_Mat];
            D27.f[dirTN  ] = &DD27[dirTN  *size_Mat];
            D27.f[dirBS  ] = &DD27[dirBS  *size_Mat];
            D27.f[dirBN  ] = &DD27[dirBN  *size_Mat];
            D27.f[dirTS  ] = &DD27[dirTS  *size_Mat];
            D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
            D27.f[dirTNE ] = &DD27[dirTNE *size_Mat];
            D27.f[dirTSW ] = &DD27[dirTSW *size_Mat];
            D27.f[dirTSE ] = &DD27[dirTSE *size_Mat];
            D27.f[dirTNW ] = &DD27[dirTNW *size_Mat];
            D27.f[dirBNE ] = &DD27[dirBNE *size_Mat];
            D27.f[dirBSW ] = &DD27[dirBSW *size_Mat];
            D27.f[dirBSE ] = &DD27[dirBSE *size_Mat];
            D27.f[dirBNW ] = &DD27[dirBNW *size_Mat];
         }
         else
         {
            D27.f[dirW   ] = &DD27[dirE   *size_Mat];
            D27.f[dirE   ] = &DD27[dirW   *size_Mat];
            D27.f[dirS   ] = &DD27[dirN   *size_Mat];
            D27.f[dirN   ] = &DD27[dirS   *size_Mat];
            D27.f[dirB   ] = &DD27[dirT   *size_Mat];
            D27.f[dirT   ] = &DD27[dirB   *size_Mat];
            D27.f[dirSW  ] = &DD27[dirNE  *size_Mat];
            D27.f[dirNE  ] = &DD27[dirSW  *size_Mat];
            D27.f[dirNW  ] = &DD27[dirSE  *size_Mat];
            D27.f[dirSE  ] = &DD27[dirNW  *size_Mat];
            D27.f[dirBW  ] = &DD27[dirTE  *size_Mat];
            D27.f[dirTE  ] = &DD27[dirBW  *size_Mat];
            D27.f[dirTW  ] = &DD27[dirBE  *size_Mat];
            D27.f[dirBE  ] = &DD27[dirTW  *size_Mat];
            D27.f[dirBS  ] = &DD27[dirTN  *size_Mat];
            D27.f[dirTN  ] = &DD27[dirBS  *size_Mat];
            D27.f[dirTS  ] = &DD27[dirBN  *size_Mat];
            D27.f[dirBN  ] = &DD27[dirTS  *size_Mat];
            D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
            D27.f[dirBSW ] = &DD27[dirTNE *size_Mat];
            D27.f[dirBNE ] = &DD27[dirTSW *size_Mat];
            D27.f[dirBNW ] = &DD27[dirTSE *size_Mat];
            D27.f[dirBSE ] = &DD27[dirTNW *size_Mat];
            D27.f[dirTSW ] = &DD27[dirBNE *size_Mat];
            D27.f[dirTNE ] = &DD27[dirBSW *size_Mat];
            D27.f[dirTNW ] = &DD27[dirBSE *size_Mat];
            D27.f[dirTSE ] = &DD27[dirBNW *size_Mat];
         }

         ////////////////////////////////////////////////////////////////////////////////
         //index
         unsigned int kw   = neighborX[k];
         unsigned int ks   = neighborY[k];
         unsigned int kb   = neighborZ[k];
         unsigned int ksw  = neighborY[kw];
         unsigned int kbw  = neighborZ[kw];
         unsigned int kbs  = neighborZ[ks];
         unsigned int kbsw = neighborZ[ksw];
         //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         doubflo fW    =  (D.f[dirE   ])[k  ];//ke
         doubflo fE    =  (D.f[dirW   ])[kw ];
         doubflo fS    =  (D.f[dirN   ])[k  ];//kn
         doubflo fN    =  (D.f[dirS   ])[ks ];
         doubflo fB    =  (D.f[dirT   ])[k  ];//kt
         doubflo fT    =  (D.f[dirB   ])[kb ];
         doubflo fSW   =  (D.f[dirNE  ])[k  ];//kne
         doubflo fNE   =  (D.f[dirSW  ])[ksw];
         doubflo fNW   =  (D.f[dirSE  ])[ks ];//kse
         doubflo fSE   =  (D.f[dirNW  ])[kw ];//knw
         doubflo fBW   =  (D.f[dirTE  ])[k  ];//kte
         doubflo fTE   =  (D.f[dirBW  ])[kbw];
         doubflo fTW   =  (D.f[dirBE  ])[kb ];//kbe
         doubflo fBE   =  (D.f[dirTW  ])[kw ];//ktw
         doubflo fBS   =  (D.f[dirTN  ])[k  ];//ktn
         doubflo fTN   =  (D.f[dirBS  ])[kbs];
         doubflo fTS   =  (D.f[dirBN  ])[kb ];//kbn
         doubflo fBN   =  (D.f[dirTS  ])[ks ];//kts
         doubflo fZERO =  (D.f[dirZERO])[k  ];//kzero
         doubflo fBSW  =  (D.f[dirTNE ])[k  ];//ktne
         doubflo fBNE  =  (D.f[dirTSW ])[ksw];//ktsw
         doubflo fBNW  =  (D.f[dirTSE ])[ks ];//ktse
         doubflo fBSE  =  (D.f[dirTNW ])[kw ];//ktnw
         doubflo fTSW  =  (D.f[dirBNE ])[kb ];//kbne
         doubflo fTNE  =  (D.f[dirBSW ])[kbsw];
         doubflo fTNW  =  (D.f[dirBSE ])[kbs];//kbse
         doubflo fTSE  =  (D.f[dirBNW ])[kbw];//kbnw
         ////////////////////////////////////////////////////////////////////////////////
			doubflo mfcbb = (D27.f[dirE   ])[k  ];
			doubflo mfabb = (D27.f[dirW   ])[kw ];
			doubflo mfbcb = (D27.f[dirN   ])[k  ];
			doubflo mfbab = (D27.f[dirS   ])[ks ];
			doubflo mfbbc = (D27.f[dirT   ])[k  ];
			doubflo mfbba = (D27.f[dirB   ])[kb ];
			doubflo mfccb = (D27.f[dirNE  ])[k  ];
			doubflo mfaab = (D27.f[dirSW  ])[ksw];
			doubflo mfcab = (D27.f[dirSE  ])[ks ];
			doubflo mfacb = (D27.f[dirNW  ])[kw ];
			doubflo mfcbc = (D27.f[dirTE  ])[k  ];
			doubflo mfaba = (D27.f[dirBW  ])[kbw];
			doubflo mfcba = (D27.f[dirBE  ])[kb ];
			doubflo mfabc = (D27.f[dirTW  ])[kw ];
			doubflo mfbcc = (D27.f[dirTN  ])[k  ];
			doubflo mfbaa = (D27.f[dirBS  ])[kbs];
			doubflo mfbca = (D27.f[dirBN  ])[kb ];
			doubflo mfbac = (D27.f[dirTS  ])[ks ];
			doubflo mfbbb = (D27.f[dirZERO])[k  ];
			doubflo mfccc = (D27.f[dirTNE ])[k  ];
			doubflo mfaac = (D27.f[dirTSW ])[ksw];
			doubflo mfcac = (D27.f[dirTSE ])[ks ];
			doubflo mfacc = (D27.f[dirTNW ])[kw ];
			doubflo mfcca = (D27.f[dirBNE ])[kb ];
			doubflo mfaaa = (D27.f[dirBSW ])[kbsw];
			doubflo mfcaa = (D27.f[dirBSE ])[kbs];
			doubflo mfaca = (D27.f[dirBNW ])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			//Conc
			doubflo drho = ((((mfccc+mfaaa) + (mfaca+mfcac)) + ((mfacc+mfcaa) + (mfaac+mfcca))) + 
							(((mfbac+mfbca) + (mfbaa+mfbcc)) + ((mfabc+mfcba) + (mfaba+mfcbc)) + ((mfacb+mfcab) + (mfaab+mfccb))) +
							((mfabb+mfcbb) + (mfbab+mfbcb)) + (mfbba+mfbbc)) + mfbbb;

			doubflo rho = one+drho;
			////////////////////////////////////////////////////////////////////////////////
			doubflo rho0fluid   =  (fTNE+fBSW)+(fTSW+fBNE)+(fTSE+fBNW)+(fTNW+fBSE)+(fNE+fSW)+(fNW+fSE)+(fTE+fBW)+(fBE+fTW)+(fTN+fBS)+(fBN+fTS)+(fE+fW)+(fN+fS)+(fT+fB)+fZERO;
			doubflo rhofluid    =  rho0fluid + one;
			doubflo OORhofluid  =  one/rhofluid;
			doubflo vvx     =  OORhofluid*((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
			doubflo vvy     =  OORhofluid*((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
			doubflo vvz     =  OORhofluid*((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
			////////////////////////////////////////////////////////////////////////////////
			//doubflo omegaD     = zero;
			doubflo omegaD     = two / (six * diffusivity + one);

			doubflo oMdrho = zero;//one; // comp special
			doubflo m0, m1, m2;
			doubflo vx2=vvx*vvx;
			doubflo vy2=vvy*vvy;
			doubflo vz2=vvz*vvz;

			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m2    = mfaaa	+ mfaac;
			m1    = mfaac	- mfaaa;
			m0    = m2		+ mfaab;
			mfaaa = m0;
			m0   += c1o36 * oMdrho;	
			mfaab = m1 -		m0 * vvz;
			mfaac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfabc;
			m1    = mfabc  - mfaba;
			m0    = m2		+ mfabb;
			mfaba = m0;
			m0   += c1o9 * oMdrho;
			mfabb = m1 -		m0 * vvz;
			mfabc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfacc;
			m1    = mfacc  - mfaca;
			m0    = m2		+ mfacb;
			mfaca = m0;
			m0   += c1o36 * oMdrho;
			mfacb = m1 -		m0 * vvz;
			mfacc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbac;
			m1    = mfbac	- mfbaa;
			m0    = m2		+ mfbab;
			mfbaa = m0;
			m0   += c1o9 * oMdrho;
			mfbab = m1 -		m0 * vvz;
			mfbac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbba  + mfbbc;
			m1    = mfbbc  - mfbba;
			m0    = m2		+ mfbbb;
			mfbba = m0;
			m0   += c4o9 * oMdrho;
			mfbbb = m1 -		m0 * vvz;
			mfbbc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbca  + mfbcc;
			m1    = mfbcc  - mfbca;
			m0    = m2		+ mfbcb;
			mfbca = m0;
			m0   += c1o9 * oMdrho;
			mfbcb = m1 -		m0 * vvz;
			mfbcc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcac;
			m1    = mfcac	- mfcaa;
			m0    = m2		+ mfcab;
			mfcaa = m0;
			m0   += c1o36 * oMdrho;
			mfcab = m1 -		m0 * vvz;
			mfcac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcba  + mfcbc;
			m1    = mfcbc  - mfcba;
			m0    = m2		+ mfcbb;
			mfcba = m0;
			m0   += c1o9 * oMdrho;
			mfcbb = m1 -		m0 * vvz;
			mfcbc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcca  + mfccc;
			m1    = mfccc  - mfcca;
			m0    = m2		+ mfccb;
			mfcca = m0;
			m0   += c1o36 * oMdrho;
			mfccb = m1 -		m0 * vvz;
			mfccc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2    = mfaaa	+ mfaca;
			m1    = mfaca	- mfaaa;
			m0    = m2		+ mfaba;
			mfaaa = m0;
			m0   += c1o6 * oMdrho;
			mfaba = m1 -		m0 * vvy;
			mfaca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab  + mfacb;
			m1    = mfacb  - mfaab;
			m0    = m2		+ mfabb;
			mfaab = m0;
			mfabb = m1 -		m0 * vvy;
			mfacb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac  + mfacc;
			m1    = mfacc  - mfaac;
			m0    = m2		+ mfabc;
			mfaac = m0;
			m0   += c1o18 * oMdrho;
			mfabc = m1 -		m0 * vvy;
			mfacc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbca;
			m1    = mfbca	- mfbaa;
			m0    = m2		+ mfbba;
			mfbaa = m0;
			m0   += c2o3 * oMdrho;
			mfbba = m1 -		m0 * vvy;
			mfbca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbab  + mfbcb;
			m1    = mfbcb  - mfbab;
			m0    = m2		+ mfbbb;
			mfbab = m0;
			mfbbb = m1 -		m0 * vvy;
			mfbcb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbac  + mfbcc;
			m1    = mfbcc  - mfbac;
			m0    = m2		+ mfbbc;
			mfbac = m0;
			m0   += c2o9 * oMdrho;
			mfbbc = m1 -		m0 * vvy;
			mfbcc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcca;
			m1    = mfcca	- mfcaa;
			m0    = m2		+ mfcba;
			mfcaa = m0;
			m0   += c1o6 * oMdrho;
			mfcba = m1 -		m0 * vvy;
			mfcca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcab  + mfccb;
			m1    = mfccb  - mfcab;
			m0    = m2		+ mfcbb;
			mfcab = m0;
			mfcbb = m1 -		m0 * vvy;
			mfccb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcac  + mfccc;
			m1    = mfccc  - mfcac;
			m0    = m2		+ mfcbc;
			mfcac = m0;
			m0   += c1o18 * oMdrho;
			mfcbc = m1 -		m0 * vvy;
			mfccc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2    = mfaaa	+ mfcaa;
			m1    = mfcaa	- mfaaa;
			m0    = m2		+ mfbaa;
			mfaaa = m0;
			m0   += one* oMdrho;
			mfbaa = m1 -		m0 * vvx;
			mfcaa = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfcba;
			m1    = mfcba  - mfaba;
			m0    = m2		+ mfbba;
			mfaba = m0;
			mfbba = m1 -		m0 * vvx;
			mfcba = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfcca;
			m1    = mfcca  - mfaca;
			m0    = m2		+ mfbca;
			mfaca = m0;
			m0   += c1o3 * oMdrho;
			mfbca = m1 -		m0 * vvx;
			mfcca = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab	+ mfcab;
			m1    = mfcab	- mfaab;
			m0    = m2		+ mfbab;
			mfaab = m0;
			mfbab = m1 -		m0 * vvx;
			mfcab = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabb  + mfcbb;
			m1    = mfcbb  - mfabb;
			m0    = m2		+ mfbbb;
			mfabb = m0;
			mfbbb = m1 -		m0 * vvx;
			mfcbb = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacb  + mfccb;
			m1    = mfccb  - mfacb;
			m0    = m2		+ mfbcb;
			mfacb = m0;
			mfbcb = m1 -		m0 * vvx;
			mfccb = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac	+ mfcac;
			m1    = mfcac	- mfaac;
			m0    = m2		+ mfbac;
			mfaac = m0;
			m0   += c1o3 * oMdrho;
			mfbac = m1 -		m0 * vvx;
			mfcac = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabc  + mfcbc;
			m1    = mfcbc  - mfabc;
			m0    = m2		+ mfbbc;
			mfabc = m0;
			mfbbc = m1 -		m0 * vvx;
			mfcbc = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacc  + mfccc;
			m1    = mfccc  - mfacc;
			m0    = m2		+ mfbcc;
			mfacc = m0;
			m0   += c1o9 * oMdrho;
			mfbcc = m1 -		m0 * vvx;
			mfccc = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////

			//if(mfaaa < zero) omegaD = one;
			doubflo limit=nine*omegaD*omegaD*(mfbaa*mfbaa+mfaba*mfaba+mfaab*mfaab);
			//doubflo CC=c1o2;
			//if ((two*mfaaa*mfaaa<limit)) omegaD=two / (six * (diffusivity+((limit/(1.0e-10f+two*mfaaa*mfaaa)-one)*(c1o6-diffusivity))*c1o2) + one);
			if ((two*mfaaa*mfaaa<limit)) omegaD=one;
			//omegaD = two / (six * (diffusivity+CC*limit) + one);

			//mfaaa = c1o2;
			//trans 3.
			doubflo Mabc=mfabc-mfaba*c1o3;
			doubflo Mbca=mfbca-mfbaa*c1o3;
			doubflo Macb=mfacb-mfaab*c1o3;
			doubflo Mcba=mfcba-mfaba*c1o3;
			doubflo Mcab=mfcab-mfaab*c1o3;
			doubflo Mbac=mfbac-mfbaa*c1o3;
			//trans 5.
			doubflo Mcbc=mfcbc-mfaba*c1o9;
			doubflo Mbcc=mfbcc-mfbaa*c1o9;
			doubflo Mccb=mfccb-mfaab*c1o9;

			//1.
			mfbaa *= one - omegaD;
			mfaba *= one - omegaD;
			mfaab *= one - omegaD;

			//3.
			//mfbca *= one - omegaD;
			//mfbac *= one - omegaD;
			//mfcba *= one - omegaD;
			//mfabc *= one - omegaD;
			//mfcab *= one - omegaD;
			//mfacb *= one - omegaD;

			//mfbbb *= one - omegaD; 
			Mabc  = zero;
			Mbca  = zero;
			Macb  = zero;
			Mcba  = zero;
			Mcab  = zero;
			Mbac  = zero;
			mfbbb = zero;

			//5.
			//mfbcc *= one - omegaD;
			//mfcbc *= one - omegaD;
			//mfccb *= one - omegaD;
			Mcbc = zero;
			Mbcc = zero;
			Mccb = zero;

			//2.
			mfbba = zero;
			mfbab = zero;
			mfabb = zero;

			mfcaa = c1o3 * drho;
			mfaca = c1o3 * drho;
			mfaac = c1o3 * drho;

			//4.
			mfacc = c1o9 * drho;
			mfcac = c1o9 * drho;
			mfcca = c1o9 * drho;

			mfcbb = zero;
			mfbcb = zero;
			mfbbc = zero;

			//6.
			mfccc = c1o27 * drho;

			//3.
			mfabc=Mabc+mfaba*c1o3;
			mfbca=Mbca+mfbaa*c1o3;
			mfacb=Macb+mfaab*c1o3;
			mfcba=Mcba+mfaba*c1o3;
			mfcab=Mcab+mfaab*c1o3;
			mfbac=Mbac+mfbaa*c1o3;
			//5.	  
			mfcbc=Mcbc+mfaba*c1o9;
			mfbcc=Mbcc+mfbaa*c1o9;
			mfccb=Mccb+mfaab*c1o9;

			////////////////////////////////////////////////////////////////////////////////////
			//back
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m0 =  mfaac * c1o2 +      mfaab * (vvz - c1o2) + (mfaaa + one* oMdrho) * (     vz2 - vvz) * c1o2; 
			m1 = -mfaac        - two* mfaab *  vvz         +  mfaaa                * (one- vz2)              - one* oMdrho * vz2; 
			m2 =  mfaac * c1o2 +      mfaab * (vvz + c1o2) + (mfaaa + one* oMdrho) * (     vz2 + vvz) * c1o2;
			mfaaa = m0;
			mfaab = m1;
			mfaac = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfabc * c1o2 +      mfabb * (vvz - c1o2) + mfaba * (     vz2 - vvz) * c1o2; 
			m1 = -mfabc        - two* mfabb *  vvz         + mfaba * (one- vz2); 
			m2 =  mfabc * c1o2 +      mfabb * (vvz + c1o2) + mfaba * (     vz2 + vvz) * c1o2;
			mfaba = m0;
			mfabb = m1;
			mfabc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfacc * c1o2 +      mfacb * (vvz - c1o2) + (mfaca + c1o3 * oMdrho) * (     vz2 - vvz) * c1o2; 
			m1 = -mfacc        - two* mfacb *  vvz         +  mfaca                  * (one- vz2)              - c1o3 * oMdrho * vz2; 
			m2 =  mfacc * c1o2 +      mfacb * (vvz + c1o2) + (mfaca + c1o3 * oMdrho) * (     vz2 + vvz) * c1o2;
			mfaca = m0;
			mfacb = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfbac * c1o2 +      mfbab * (vvz - c1o2) + mfbaa * (     vz2 - vvz) * c1o2; 
			m1 = -mfbac        - two* mfbab *  vvz         + mfbaa * (one- vz2); 
			m2 =  mfbac * c1o2 +      mfbab * (vvz + c1o2) + mfbaa * (     vz2 + vvz) * c1o2;
			mfbaa = m0;
			mfbab = m1;
			mfbac = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 =  mfbbc * c1o2 +      mfbbb * (vvz - c1o2) + mfbba * (     vz2 - vvz) * c1o2; 
			m1 = -mfbbc        - two* mfbbb *  vvz         + mfbba * (one- vz2); 
			m2 =  mfbbc * c1o2 +      mfbbb * (vvz + c1o2) + mfbba * (     vz2 + vvz) * c1o2;
			mfbba = m0;
			mfbbb = m1;
			mfbbc = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 =  mfbcc * c1o2 +      mfbcb * (vvz - c1o2) + mfbca * (     vz2 - vvz) * c1o2; 
			m1 = -mfbcc        - two* mfbcb *  vvz         + mfbca * (one- vz2); 
			m2 =  mfbcc * c1o2 +      mfbcb * (vvz + c1o2) + mfbca * (     vz2 + vvz) * c1o2;
			mfbca = m0;
			mfbcb = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcac * c1o2 +      mfcab * (vvz - c1o2) + (mfcaa + c1o3 * oMdrho) * (     vz2 - vvz) * c1o2; 
			m1 = -mfcac        - two* mfcab *  vvz         +  mfcaa                  * (one- vz2)              - c1o3 * oMdrho * vz2; 
			m2 =  mfcac * c1o2 +      mfcab * (vvz + c1o2) + (mfcaa + c1o3 * oMdrho) * (     vz2 + vvz) * c1o2;
			mfcaa = m0;
			mfcab = m1;
			mfcac = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 =  mfcbc * c1o2 +      mfcbb * (vvz - c1o2) + mfcba * (     vz2 - vvz) * c1o2; 
			m1 = -mfcbc        - two* mfcbb *  vvz         + mfcba * (one- vz2); 
			m2 =  mfcbc * c1o2 +      mfcbb * (vvz + c1o2) + mfcba * (     vz2 + vvz) * c1o2;
			mfcba = m0;
			mfcbb = m1;
			mfcbc = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 =  mfccc * c1o2 +      mfccb * (vvz - c1o2) + (mfcca + c1o9 * oMdrho) * (     vz2 - vvz) * c1o2; 
			m1 = -mfccc        - two* mfccb *  vvz         +  mfcca                  * (one- vz2)              - c1o9 * oMdrho * vz2; 
			m2 =  mfccc * c1o2 +      mfccb * (vvz + c1o2) + (mfcca + c1o9 * oMdrho) * (     vz2 + vvz) * c1o2;
			mfcca = m0;
			mfccb = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m0 =  mfaca * c1o2 +      mfaba * (vvy - c1o2) + (mfaaa + c1o6 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfaca        - two* mfaba *  vvy         +  mfaaa                  * (one- vy2)              - c1o6 * oMdrho * vy2; 
			m2 =  mfaca * c1o2 +      mfaba * (vvy + c1o2) + (mfaaa + c1o6 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfaaa = m0;
			mfaba = m1;
			mfaca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfacb * c1o2 +      mfabb * (vvy - c1o2) + (mfaab + c2o3 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfacb        - two* mfabb *  vvy         +  mfaab                  * (one- vy2)              - c2o3 * oMdrho * vy2; 
			m2 =  mfacb * c1o2 +      mfabb * (vvy + c1o2) + (mfaab + c2o3 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfaab = m0;
			mfabb = m1;
			mfacb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfacc * c1o2 +      mfabc * (vvy - c1o2) + (mfaac + c1o6 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfacc        - two* mfabc *  vvy         +  mfaac                  * (one- vy2)              - c1o6 * oMdrho * vy2; 
			m2 =  mfacc * c1o2 +      mfabc * (vvy + c1o2) + (mfaac + c1o6 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfaac = m0;
			mfabc = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfbca * c1o2 +      mfbba * (vvy - c1o2) + mfbaa * (     vy2 - vvy) * c1o2; 
			m1 = -mfbca        - two* mfbba *  vvy         + mfbaa * (one- vy2); 
			m2 =  mfbca * c1o2 +      mfbba * (vvy + c1o2) + mfbaa * (     vy2 + vvy) * c1o2;
			mfbaa = m0;
			mfbba = m1;
			mfbca = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 =  mfbcb * c1o2 +      mfbbb * (vvy - c1o2) + mfbab * (     vy2 - vvy) * c1o2; 
			m1 = -mfbcb        - two* mfbbb *  vvy         + mfbab * (one- vy2); 
			m2 =  mfbcb * c1o2 +      mfbbb * (vvy + c1o2) + mfbab * (     vy2 + vvy) * c1o2;
			mfbab = m0;
			mfbbb = m1;
			mfbcb = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 =  mfbcc * c1o2 +      mfbbc * (vvy - c1o2) + mfbac * (     vy2 - vvy) * c1o2; 
			m1 = -mfbcc        - two* mfbbc *  vvy         + mfbac * (one- vy2); 
			m2 =  mfbcc * c1o2 +      mfbbc * (vvy + c1o2) + mfbac * (     vy2 + vvy) * c1o2;
			mfbac = m0;
			mfbbc = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcca * c1o2 +      mfcba * (vvy - c1o2) + (mfcaa + c1o18 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfcca        - two* mfcba *  vvy         +  mfcaa                   * (one- vy2)              - c1o18 * oMdrho * vy2; 
			m2 =  mfcca * c1o2 +      mfcba * (vvy + c1o2) + (mfcaa + c1o18 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfcaa = m0;
			mfcba = m1;
			mfcca = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 =  mfccb * c1o2 +      mfcbb * (vvy - c1o2) + (mfcab + c2o9 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfccb        - two* mfcbb *  vvy         +  mfcab                  * (one- vy2)              - c2o9 * oMdrho * vy2; 
			m2 =  mfccb * c1o2 +      mfcbb * (vvy + c1o2) + (mfcab + c2o9 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfcab = m0;
			mfcbb = m1;
			mfccb = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 =  mfccc * c1o2 +      mfcbc * (vvy - c1o2) + (mfcac + c1o18 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfccc        - two* mfcbc *  vvy         +  mfcac                   * (one- vy2)              - c1o18 * oMdrho * vy2; 
			m2 =  mfccc * c1o2 +      mfcbc * (vvy + c1o2) + (mfcac + c1o18 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfcac = m0;
			mfcbc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m0 =  mfcaa * c1o2 +      mfbaa * (vvx - c1o2) + (mfaaa + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcaa        - two* mfbaa *  vvx         +  mfaaa                   * (one- vx2)              - c1o36 * oMdrho * vx2; 
			m2 =  mfcaa * c1o2 +      mfbaa * (vvx + c1o2) + (mfaaa + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaaa = m0;
			mfbaa = m1;
			mfcaa = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcba * c1o2 +      mfbba * (vvx - c1o2) + (mfaba + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcba        - two* mfbba *  vvx         +  mfaba                  * (one- vx2)              - c1o9 * oMdrho * vx2; 
			m2 =  mfcba * c1o2 +      mfbba * (vvx + c1o2) + (mfaba + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaba = m0;
			mfbba = m1;
			mfcba = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcca * c1o2 +      mfbca * (vvx - c1o2) + (mfaca + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcca        - two* mfbca *  vvx         +  mfaca                   * (one- vx2)              - c1o36 * oMdrho * vx2; 
			m2 =  mfcca * c1o2 +      mfbca * (vvx + c1o2) + (mfaca + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaca = m0;
			mfbca = m1;
			mfcca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcab * c1o2 +      mfbab * (vvx - c1o2) + (mfaab + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcab        - two* mfbab *  vvx         +  mfaab                  * (one- vx2)              - c1o9 * oMdrho * vx2; 
			m2 =  mfcab * c1o2 +      mfbab * (vvx + c1o2) + (mfaab + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaab = m0;
			mfbab = m1;
			mfcab = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 =  mfcbb * c1o2 +      mfbbb * (vvx - c1o2) + (mfabb + c4o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcbb        - two* mfbbb *  vvx         +  mfabb                  * (one- vx2)              - c4o9 * oMdrho * vx2; 
			m2 =  mfcbb * c1o2 +      mfbbb * (vvx + c1o2) + (mfabb + c4o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfabb = m0;
			mfbbb = m1;
			mfcbb = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 =  mfccb * c1o2 +      mfbcb * (vvx - c1o2) + (mfacb + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfccb        - two* mfbcb *  vvx         +  mfacb                  * (one- vx2)              - c1o9 * oMdrho * vx2; 
			m2 =  mfccb * c1o2 +      mfbcb * (vvx + c1o2) + (mfacb + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfacb = m0;
			mfbcb = m1;
			mfccb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcac * c1o2 +      mfbac * (vvx - c1o2) + (mfaac + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcac        - two* mfbac *  vvx         +  mfaac                   * (one- vx2)              - c1o36 * oMdrho * vx2; 
			m2 =  mfcac * c1o2 +      mfbac * (vvx + c1o2) + (mfaac + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaac = m0;
			mfbac = m1;
			mfcac = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 =  mfcbc * c1o2 +      mfbbc * (vvx - c1o2) + (mfabc + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcbc        - two* mfbbc *  vvx         +  mfabc                  * (one- vx2)              - c1o9 * oMdrho * vx2; 
			m2 =  mfcbc * c1o2 +      mfbbc * (vvx + c1o2) + (mfabc + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfabc = m0;
			mfbbc = m1;
			mfcbc = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 =  mfccc * c1o2 +      mfbcc * (vvx - c1o2) + (mfacc + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfccc        - two* mfbcc *  vvx         +  mfacc                   * (one- vx2)              - c1o36 * oMdrho * vx2; 
			m2 =  mfccc * c1o2 +      mfbcc * (vvx + c1o2) + (mfacc + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfacc = m0;
			mfbcc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			(D27.f[ dirE   ])[k   ] = mfabb;                                                                   
			(D27.f[ dirW   ])[kw  ] = mfcbb;                                                                 
			(D27.f[ dirN   ])[k   ] = mfbab;
			(D27.f[ dirS   ])[ks  ] = mfbcb;
			(D27.f[ dirT   ])[k   ] = mfbba;
			(D27.f[ dirB   ])[kb  ] = mfbbc;
			(D27.f[ dirNE  ])[k   ] = mfaab;
			(D27.f[ dirSW  ])[ksw ] = mfccb;
			(D27.f[ dirSE  ])[ks  ] = mfacb;
			(D27.f[ dirNW  ])[kw  ] = mfcab;
			(D27.f[ dirTE  ])[k   ] = mfaba;
			(D27.f[ dirBW  ])[kbw ] = mfcbc;
			(D27.f[ dirBE  ])[kb  ] = mfabc;
			(D27.f[ dirTW  ])[kw  ] = mfcba;
			(D27.f[ dirTN  ])[k   ] = mfbaa;
			(D27.f[ dirBS  ])[kbs ] = mfbcc;
			(D27.f[ dirBN  ])[kb  ] = mfbac;
			(D27.f[ dirTS  ])[ks  ] = mfbca;
			(D27.f[ dirZERO])[k   ] = mfbbb;
			(D27.f[ dirTNE ])[k   ] = mfaaa;
			(D27.f[ dirTSE ])[ks  ] = mfaca;
			(D27.f[ dirBNE ])[kb  ] = mfaac;
			(D27.f[ dirBSE ])[kbs ] = mfacc;
			(D27.f[ dirTNW ])[kw  ] = mfcaa;
			(D27.f[ dirTSW ])[ksw ] = mfcca;
			(D27.f[ dirBNW ])[kbw ] = mfcac;
			(D27.f[ dirBSW ])[kbsw] = mfccc;
			////////////////////////////////////////////////////////////////////////////////////
      }                                                                                                                    
   }
}


























////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LB_Kernel_ThS7( doubflo diffusivity,
                                           unsigned int* bcMatD,
                                           unsigned int* neighborX,
                                           unsigned int* neighborY,
                                           unsigned int* neighborZ,
                                           doubflo* DDStart,
                                           doubflo* DD7,
                                           int size_Mat,
                                           bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   bcMatD[k];

      if( (BC != GEO_SOLID) && (BC != GEO_VOID))
      {
         Distributions27 D;
         if (EvenOrOdd==true)
         {
            D.f[dirE   ] = &DDStart[dirE   *size_Mat];
            D.f[dirW   ] = &DDStart[dirW   *size_Mat];
            D.f[dirN   ] = &DDStart[dirN   *size_Mat];
            D.f[dirS   ] = &DDStart[dirS   *size_Mat];
            D.f[dirT   ] = &DDStart[dirT   *size_Mat];
            D.f[dirB   ] = &DDStart[dirB   *size_Mat];
            D.f[dirNE  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirSW  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirTNE ] = &DDStart[dirTNE *size_Mat];
            D.f[dirTSW ] = &DDStart[dirTSW *size_Mat];
            D.f[dirTSE ] = &DDStart[dirTSE *size_Mat];
            D.f[dirTNW ] = &DDStart[dirTNW *size_Mat];
            D.f[dirBNE ] = &DDStart[dirBNE *size_Mat];
            D.f[dirBSW ] = &DDStart[dirBSW *size_Mat];
            D.f[dirBSE ] = &DDStart[dirBSE *size_Mat];
            D.f[dirBNW ] = &DDStart[dirBNW *size_Mat];
         }
         else
         {
            D.f[dirW   ] = &DDStart[dirE   *size_Mat];
            D.f[dirE   ] = &DDStart[dirW   *size_Mat];
            D.f[dirS   ] = &DDStart[dirN   *size_Mat];
            D.f[dirN   ] = &DDStart[dirS   *size_Mat];
            D.f[dirB   ] = &DDStart[dirT   *size_Mat];
            D.f[dirT   ] = &DDStart[dirB   *size_Mat];
            D.f[dirSW  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirNE  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirBSW ] = &DDStart[dirTNE *size_Mat];
            D.f[dirBNE ] = &DDStart[dirTSW *size_Mat];
            D.f[dirBNW ] = &DDStart[dirTSE *size_Mat];
            D.f[dirBSE ] = &DDStart[dirTNW *size_Mat];
            D.f[dirTSW ] = &DDStart[dirBNE *size_Mat];
            D.f[dirTNE ] = &DDStart[dirBSW *size_Mat];
            D.f[dirTNW ] = &DDStart[dirBSE *size_Mat];
            D.f[dirTSE ] = &DDStart[dirBNW *size_Mat];
         }

         Distributions7 D7;
         if (EvenOrOdd==true)
         {
            D7.f[0] = &DD7[0*size_Mat];
            D7.f[1] = &DD7[1*size_Mat];
            D7.f[2] = &DD7[2*size_Mat];
            D7.f[3] = &DD7[3*size_Mat];
            D7.f[4] = &DD7[4*size_Mat];
            D7.f[5] = &DD7[5*size_Mat];
            D7.f[6] = &DD7[6*size_Mat];
         }
         else
         {
            D7.f[0] = &DD7[0*size_Mat];
            D7.f[2] = &DD7[1*size_Mat];
            D7.f[1] = &DD7[2*size_Mat];
            D7.f[4] = &DD7[3*size_Mat];
            D7.f[3] = &DD7[4*size_Mat];
            D7.f[6] = &DD7[5*size_Mat];
            D7.f[5] = &DD7[6*size_Mat];
         }

         ////////////////////////////////////////////////////////////////////////////////
         //index
         unsigned int kw   = neighborX[k];
         unsigned int ks   = neighborY[k];
         unsigned int kb   = neighborZ[k];
         unsigned int ksw  = neighborY[kw];
         unsigned int kbw  = neighborZ[kw];
         unsigned int kbs  = neighborZ[ks];
         unsigned int kbsw = neighborZ[ksw];
         //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         doubflo fW    =  (D.f[dirE   ])[k  ];//ke
         doubflo fE    =  (D.f[dirW   ])[kw ];
         doubflo fS    =  (D.f[dirN   ])[k  ];//kn
         doubflo fN    =  (D.f[dirS   ])[ks ];
         doubflo fB    =  (D.f[dirT   ])[k  ];//kt
         doubflo fT    =  (D.f[dirB   ])[kb ];
         doubflo fSW   =  (D.f[dirNE  ])[k  ];//kne
         doubflo fNE   =  (D.f[dirSW  ])[ksw];
         doubflo fNW   =  (D.f[dirSE  ])[ks ];//kse
         doubflo fSE   =  (D.f[dirNW  ])[kw ];//knw
         doubflo fBW   =  (D.f[dirTE  ])[k  ];//kte
         doubflo fTE   =  (D.f[dirBW  ])[kbw];
         doubflo fTW   =  (D.f[dirBE  ])[kb ];//kbe
         doubflo fBE   =  (D.f[dirTW  ])[kw ];//ktw
         doubflo fBS   =  (D.f[dirTN  ])[k  ];//ktn
         doubflo fTN   =  (D.f[dirBS  ])[kbs];
         doubflo fTS   =  (D.f[dirBN  ])[kb ];//kbn
         doubflo fBN   =  (D.f[dirTS  ])[ks ];//kts
         doubflo fZERO =  (D.f[dirZERO])[k  ];//kzero
         doubflo fBSW   = (D.f[dirTNE ])[k  ];//ktne
         doubflo fBNE   = (D.f[dirTSW ])[ksw];//ktsw
         doubflo fBNW   = (D.f[dirTSE ])[ks ];//ktse
         doubflo fBSE   = (D.f[dirTNW ])[kw ];//ktnw
         doubflo fTSW   = (D.f[dirBNE ])[kb ];//kbne
         doubflo fTNE   = (D.f[dirBSW ])[kbsw];
         doubflo fTNW   = (D.f[dirBSE ])[kbs];//kbse
         doubflo fTSE   = (D.f[dirBNW ])[kbw];//kbnw
         //doubflo fE    =  (D.f[dirE   ])[k  ];//ke
         //doubflo fW    =  (D.f[dirW   ])[kw ];
         //doubflo fN    =  (D.f[dirN   ])[k  ];//kn
         //doubflo fS    =  (D.f[dirS   ])[ks ];
         //doubflo fT    =  (D.f[dirT   ])[k  ];//kt
         //doubflo fB    =  (D.f[dirB   ])[kb ];
         //doubflo fNE   =  (D.f[dirNE  ])[k  ];//kne
         //doubflo fSW   =  (D.f[dirSW  ])[ksw];
         //doubflo fSE   =  (D.f[dirSE  ])[ks ];//kse
         //doubflo fNW   =  (D.f[dirNW  ])[kw ];//knw
         //doubflo fTE   =  (D.f[dirTE  ])[k  ];//kte
         //doubflo fBW   =  (D.f[dirBW  ])[kbw];
         //doubflo fBE   =  (D.f[dirBE  ])[kb ];//kbe
         //doubflo fTW   =  (D.f[dirTW  ])[kw ];//ktw
         //doubflo fTN   =  (D.f[dirTN  ])[k  ];//ktn
         //doubflo fBS   =  (D.f[dirBS  ])[kbs];
         //doubflo fBN   =  (D.f[dirBN  ])[kb ];//kbn
         //doubflo fTS   =  (D.f[dirTS  ])[ks ];//kts
         //doubflo fZERO =  (D.f[dirZERO])[k  ];//kzero
         //doubflo fTNE   = (D.f[dirTNE ])[k  ];//ktne
         //doubflo fTSW   = (D.f[dirTSW ])[ksw];//ktsw
         //doubflo fTSE   = (D.f[dirTSE ])[ks ];//ktse
         //doubflo fTNW   = (D.f[dirTNW ])[kw ];//ktnw
         //doubflo fBNE   = (D.f[dirBNE ])[kb ];//kbne
         //doubflo fBSW   = (D.f[dirBSW ])[kbsw];
         //doubflo fBSE   = (D.f[dirBSE ])[kbs];//kbse
         //doubflo fBNW   = (D.f[dirBNW ])[kbw];//kbnw
         ////////////////////////////////////////////////////////////////////////////////
         doubflo f7ZERO =  (D7.f[0])[k  ];
         doubflo f7E    =  (D7.f[1])[k  ];
         doubflo f7W    =  (D7.f[2])[kw ];
         doubflo f7N    =  (D7.f[3])[k  ];
         doubflo f7S    =  (D7.f[4])[ks ];
         doubflo f7T    =  (D7.f[5])[k  ];
         doubflo f7B    =  (D7.f[6])[kb ];
         ////////////////////////////////////////////////////////////////////////////////
         doubflo rho0   =  (fTNE+fBSW)+(fTSW+fBNE)+(fTSE+fBNW)+(fTNW+fBSE)+(fNE+fSW)+(fNW+fSE)+(fTE+fBW)+(fBE+fTW)+(fTN+fBS)+(fBN+fTS)+(fE+fW)+(fN+fS)+(fT+fB)+fZERO;
         doubflo rho    =  rho0 + one;
         doubflo OORho  =  one/rho;
         doubflo vx     =  OORho*((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
         doubflo vy     =  OORho*((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
         doubflo vz     =  OORho*((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
         ////////////////////////////////////////////////////////////////////////////////
         doubflo omegaD     = -three + sqrt(three);
         doubflo Lam         = -(c1o2+one/omegaD);
         doubflo nue_d       = Lam/three;
         doubflo ae          = diffusivity/nue_d - one;
         doubflo ux_sq       = vx * vx;
         doubflo uy_sq       = vy * vy;
         doubflo uz_sq       = vz * vz;

         doubflo ConcD       = f7ZERO+f7E+f7W+f7N+f7S+f7T+f7B;

         (D7.f[0])[k  ] = f7ZERO*(one+omegaD)-omegaD*ConcD*(c1o3*(ae*(-three))-(ux_sq+uy_sq+uz_sq));
         (D7.f[2])[kw ] = f7E   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)+vx*c1o2);
         (D7.f[1])[k  ] = f7W   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)-vx*c1o2);
         (D7.f[4])[ks ] = f7N   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)+vy*c1o2);
         (D7.f[3])[k  ] = f7S   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)-vy*c1o2);
         (D7.f[6])[kb ] = f7T   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)+vz*c1o2);
         (D7.f[5])[k  ] = f7B   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)-vz*c1o2);
      }                                                                                                                    
   }
}
////////////////////////////////////////////////////////////////////////////////








































////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////																																		  //////
//////                 										incomp   																		  //////
//////																																		  //////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////




////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LB_Kernel_AD_Incomp_27(   doubflo diffusivity,
													 unsigned int* bcMatD,
													 unsigned int* neighborX,
													 unsigned int* neighborY,
													 unsigned int* neighborZ,
													 doubflo* DDStart,
													 doubflo* DD27,
													 int size_Mat,
													 bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   bcMatD[k];

      if( (BC != GEO_SOLID) && (BC != GEO_VOID))
      {
         Distributions27 D;
         if (EvenOrOdd==true)
         {
            D.f[dirE   ] = &DDStart[dirE   *size_Mat];
            D.f[dirW   ] = &DDStart[dirW   *size_Mat];
            D.f[dirN   ] = &DDStart[dirN   *size_Mat];
            D.f[dirS   ] = &DDStart[dirS   *size_Mat];
            D.f[dirT   ] = &DDStart[dirT   *size_Mat];
            D.f[dirB   ] = &DDStart[dirB   *size_Mat];
            D.f[dirNE  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirSW  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirTNE ] = &DDStart[dirTNE *size_Mat];
            D.f[dirTSW ] = &DDStart[dirTSW *size_Mat];
            D.f[dirTSE ] = &DDStart[dirTSE *size_Mat];
            D.f[dirTNW ] = &DDStart[dirTNW *size_Mat];
            D.f[dirBNE ] = &DDStart[dirBNE *size_Mat];
            D.f[dirBSW ] = &DDStart[dirBSW *size_Mat];
            D.f[dirBSE ] = &DDStart[dirBSE *size_Mat];
            D.f[dirBNW ] = &DDStart[dirBNW *size_Mat];
         }
         else
         {
            D.f[dirW   ] = &DDStart[dirE   *size_Mat];
            D.f[dirE   ] = &DDStart[dirW   *size_Mat];
            D.f[dirS   ] = &DDStart[dirN   *size_Mat];
            D.f[dirN   ] = &DDStart[dirS   *size_Mat];
            D.f[dirB   ] = &DDStart[dirT   *size_Mat];
            D.f[dirT   ] = &DDStart[dirB   *size_Mat];
            D.f[dirSW  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirNE  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirBSW ] = &DDStart[dirTNE *size_Mat];
            D.f[dirBNE ] = &DDStart[dirTSW *size_Mat];
            D.f[dirBNW ] = &DDStart[dirTSE *size_Mat];
            D.f[dirBSE ] = &DDStart[dirTNW *size_Mat];
            D.f[dirTSW ] = &DDStart[dirBNE *size_Mat];
            D.f[dirTNE ] = &DDStart[dirBSW *size_Mat];
            D.f[dirTNW ] = &DDStart[dirBSE *size_Mat];
            D.f[dirTSE ] = &DDStart[dirBNW *size_Mat];
         }

         Distributions27 D27;
         if (EvenOrOdd==true)
         {
            D27.f[dirE   ] = &DD27[dirE   *size_Mat];
            D27.f[dirW   ] = &DD27[dirW   *size_Mat];
            D27.f[dirN   ] = &DD27[dirN   *size_Mat];
            D27.f[dirS   ] = &DD27[dirS   *size_Mat];
            D27.f[dirT   ] = &DD27[dirT   *size_Mat];
            D27.f[dirB   ] = &DD27[dirB   *size_Mat];
            D27.f[dirNE  ] = &DD27[dirNE  *size_Mat];
            D27.f[dirSW  ] = &DD27[dirSW  *size_Mat];
            D27.f[dirSE  ] = &DD27[dirSE  *size_Mat];
            D27.f[dirNW  ] = &DD27[dirNW  *size_Mat];
            D27.f[dirTE  ] = &DD27[dirTE  *size_Mat];
            D27.f[dirBW  ] = &DD27[dirBW  *size_Mat];
            D27.f[dirBE  ] = &DD27[dirBE  *size_Mat];
            D27.f[dirTW  ] = &DD27[dirTW  *size_Mat];
            D27.f[dirTN  ] = &DD27[dirTN  *size_Mat];
            D27.f[dirBS  ] = &DD27[dirBS  *size_Mat];
            D27.f[dirBN  ] = &DD27[dirBN  *size_Mat];
            D27.f[dirTS  ] = &DD27[dirTS  *size_Mat];
            D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
            D27.f[dirTNE ] = &DD27[dirTNE *size_Mat];
            D27.f[dirTSW ] = &DD27[dirTSW *size_Mat];
            D27.f[dirTSE ] = &DD27[dirTSE *size_Mat];
            D27.f[dirTNW ] = &DD27[dirTNW *size_Mat];
            D27.f[dirBNE ] = &DD27[dirBNE *size_Mat];
            D27.f[dirBSW ] = &DD27[dirBSW *size_Mat];
            D27.f[dirBSE ] = &DD27[dirBSE *size_Mat];
            D27.f[dirBNW ] = &DD27[dirBNW *size_Mat];
         }
         else
         {
            D27.f[dirW   ] = &DD27[dirE   *size_Mat];
            D27.f[dirE   ] = &DD27[dirW   *size_Mat];
            D27.f[dirS   ] = &DD27[dirN   *size_Mat];
            D27.f[dirN   ] = &DD27[dirS   *size_Mat];
            D27.f[dirB   ] = &DD27[dirT   *size_Mat];
            D27.f[dirT   ] = &DD27[dirB   *size_Mat];
            D27.f[dirSW  ] = &DD27[dirNE  *size_Mat];
            D27.f[dirNE  ] = &DD27[dirSW  *size_Mat];
            D27.f[dirNW  ] = &DD27[dirSE  *size_Mat];
            D27.f[dirSE  ] = &DD27[dirNW  *size_Mat];
            D27.f[dirBW  ] = &DD27[dirTE  *size_Mat];
            D27.f[dirTE  ] = &DD27[dirBW  *size_Mat];
            D27.f[dirTW  ] = &DD27[dirBE  *size_Mat];
            D27.f[dirBE  ] = &DD27[dirTW  *size_Mat];
            D27.f[dirBS  ] = &DD27[dirTN  *size_Mat];
            D27.f[dirTN  ] = &DD27[dirBS  *size_Mat];
            D27.f[dirTS  ] = &DD27[dirBN  *size_Mat];
            D27.f[dirBN  ] = &DD27[dirTS  *size_Mat];
            D27.f[dirZERO] = &DD27[dirZERO*size_Mat];
            D27.f[dirBSW ] = &DD27[dirTNE *size_Mat];
            D27.f[dirBNE ] = &DD27[dirTSW *size_Mat];
            D27.f[dirBNW ] = &DD27[dirTSE *size_Mat];
            D27.f[dirBSE ] = &DD27[dirTNW *size_Mat];
            D27.f[dirTSW ] = &DD27[dirBNE *size_Mat];
            D27.f[dirTNE ] = &DD27[dirBSW *size_Mat];
            D27.f[dirTNW ] = &DD27[dirBSE *size_Mat];
            D27.f[dirTSE ] = &DD27[dirBNW *size_Mat];
         }

         ////////////////////////////////////////////////////////////////////////////////
         //index
         unsigned int kw   = neighborX[k];
         unsigned int ks   = neighborY[k];
         unsigned int kb   = neighborZ[k];
         unsigned int ksw  = neighborY[kw];
         unsigned int kbw  = neighborZ[kw];
         unsigned int kbs  = neighborZ[ks];
         unsigned int kbsw = neighborZ[ksw];
         //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         doubflo fW    =  (D.f[dirE   ])[k  ];//ke
         doubflo fE    =  (D.f[dirW   ])[kw ];
         doubflo fS    =  (D.f[dirN   ])[k  ];//kn
         doubflo fN    =  (D.f[dirS   ])[ks ];
         doubflo fB    =  (D.f[dirT   ])[k  ];//kt
         doubflo fT    =  (D.f[dirB   ])[kb ];
         doubflo fSW   =  (D.f[dirNE  ])[k  ];//kne
         doubflo fNE   =  (D.f[dirSW  ])[ksw];
         doubflo fNW   =  (D.f[dirSE  ])[ks ];//kse
         doubflo fSE   =  (D.f[dirNW  ])[kw ];//knw
         doubflo fBW   =  (D.f[dirTE  ])[k  ];//kte
         doubflo fTE   =  (D.f[dirBW  ])[kbw];
         doubflo fTW   =  (D.f[dirBE  ])[kb ];//kbe
         doubflo fBE   =  (D.f[dirTW  ])[kw ];//ktw
         doubflo fBS   =  (D.f[dirTN  ])[k  ];//ktn
         doubflo fTN   =  (D.f[dirBS  ])[kbs];
         doubflo fTS   =  (D.f[dirBN  ])[kb ];//kbn
         doubflo fBN   =  (D.f[dirTS  ])[ks ];//kts
         doubflo fZERO =  (D.f[dirZERO])[k  ];//kzero
         doubflo fBSW  =  (D.f[dirTNE ])[k  ];//ktne
         doubflo fBNE  =  (D.f[dirTSW ])[ksw];//ktsw
         doubflo fBNW  =  (D.f[dirTSE ])[ks ];//ktse
         doubflo fBSE  =  (D.f[dirTNW ])[kw ];//ktnw
         doubflo fTSW  =  (D.f[dirBNE ])[kb ];//kbne
         doubflo fTNE  =  (D.f[dirBSW ])[kbsw];
         doubflo fTNW  =  (D.f[dirBSE ])[kbs];//kbse
         doubflo fTSE  =  (D.f[dirBNW ])[kbw];//kbnw
         ////////////////////////////////////////////////////////////////////////////////
         //doubflo f27E    =  (D27.f[dirE   ])[k  ];//ke
         //doubflo f27W    =  (D27.f[dirW   ])[kw ];
         //doubflo f27N    =  (D27.f[dirN   ])[k  ];//kn
         //doubflo f27S    =  (D27.f[dirS   ])[ks ];
         //doubflo f27T    =  (D27.f[dirT   ])[k  ];//kt
         //doubflo f27B    =  (D27.f[dirB   ])[kb ];
         //doubflo f27NE   =  (D27.f[dirNE  ])[k  ];//kne
         //doubflo f27SW   =  (D27.f[dirSW  ])[ksw];
         //doubflo f27SE   =  (D27.f[dirSE  ])[ks ];//kse
         //doubflo f27NW   =  (D27.f[dirNW  ])[kw ];//knw
         //doubflo f27TE   =  (D27.f[dirTE  ])[k  ];//kte
         //doubflo f27BW   =  (D27.f[dirBW  ])[kbw];
         //doubflo f27BE   =  (D27.f[dirBE  ])[kb ];//kbe
         //doubflo f27TW   =  (D27.f[dirTW  ])[kw ];//ktw
         //doubflo f27TN   =  (D27.f[dirTN  ])[k  ];//ktn
         //doubflo f27BS   =  (D27.f[dirBS  ])[kbs];
         //doubflo f27BN   =  (D27.f[dirBN  ])[kb ];//kbn
         //doubflo f27TS   =  (D27.f[dirTS  ])[ks ];//kts
         //doubflo f27ZERO =  (D27.f[dirZERO])[k  ];//kzero
         //doubflo f27TNE  =  (D27.f[dirTNE ])[k  ];//ktne
         //doubflo f27TSW  =  (D27.f[dirTSW ])[ksw];//ktsw
         //doubflo f27TSE  =  (D27.f[dirTSE ])[ks ];//ktse
         //doubflo f27TNW  =  (D27.f[dirTNW ])[kw ];//ktnw
         //doubflo f27BNE  =  (D27.f[dirBNE ])[kb ];//kbne
         //doubflo f27BSW  =  (D27.f[dirBSW ])[kbsw];
         //doubflo f27BSE  =  (D27.f[dirBSE ])[kbs];//kbse
         //doubflo f27BNW  =  (D27.f[dirBNW ])[kbw];//kbnw
         ////////////////////////////////////////////////////////////////////////////////
         //doubflo vx1     =  ((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
         //doubflo vx2     =  ((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
         //doubflo vx3     =  ((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
         ////////////////////////////////////////////////////////////////////////////////

		 
		 	doubflo mfcbb = (D27.f[dirE   ])[k  ];
			doubflo mfabb = (D27.f[dirW   ])[kw ];
			doubflo mfbcb = (D27.f[dirN   ])[k  ];
			doubflo mfbab = (D27.f[dirS   ])[ks ];
			doubflo mfbbc = (D27.f[dirT   ])[k  ];
			doubflo mfbba = (D27.f[dirB   ])[kb ];
			doubflo mfccb = (D27.f[dirNE  ])[k  ];
			doubflo mfaab = (D27.f[dirSW  ])[ksw];
			doubflo mfcab = (D27.f[dirSE  ])[ks ];
			doubflo mfacb = (D27.f[dirNW  ])[kw ];
			doubflo mfcbc = (D27.f[dirTE  ])[k  ];
			doubflo mfaba = (D27.f[dirBW  ])[kbw];
			doubflo mfcba = (D27.f[dirBE  ])[kb ];
			doubflo mfabc = (D27.f[dirTW  ])[kw ];
			doubflo mfbcc = (D27.f[dirTN  ])[k  ];
			doubflo mfbaa = (D27.f[dirBS  ])[kbs];
			doubflo mfbca = (D27.f[dirBN  ])[kb ];
			doubflo mfbac = (D27.f[dirTS  ])[ks ];
			doubflo mfbbb = (D27.f[dirZERO])[k  ];
			doubflo mfccc = (D27.f[dirTNE ])[k  ];
			doubflo mfaac = (D27.f[dirTSW ])[ksw];
			doubflo mfcac = (D27.f[dirTSE ])[ks ];
			doubflo mfacc = (D27.f[dirTNW ])[kw ];
			doubflo mfcca = (D27.f[dirBNE ])[kb ];
			doubflo mfaaa = (D27.f[dirBSW ])[kbsw];
			doubflo mfcaa = (D27.f[dirBSE ])[kbs];
			doubflo mfaca = (D27.f[dirBNW ])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			//Conc
			doubflo drho = ((((mfccc+mfaaa) + (mfaca+mfcac)) + ((mfacc+mfcaa) + (mfaac+mfcca))) + 
							(((mfbac+mfbca) + (mfbaa+mfbcc)) + ((mfabc+mfcba) + (mfaba+mfcbc)) + ((mfacb+mfcab) + (mfaab+mfccb))) +
							((mfabb+mfcbb) + (mfbab+mfbcb)) + (mfbba+mfbbc)) + mfbbb;
			doubflo rho = one+drho;
			////////////////////////////////////////////////////////////////////////////////////

			doubflo vvx     =  ((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
			doubflo vvy     =  ((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
			doubflo vvz     =  ((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
		 ////////////////////////////////////////////////////////////////////////////////
         doubflo omegaD     = two / (six * diffusivity + one);
         ////doubflo omegaD     = -three + sqrt(three);
         ////doubflo Lam         = -(c1o2+one/omegaD);
         ////doubflo nue_d       = Lam/three;
         //doubflo ae          = zero;
         ////doubflo ae          = diffusivity/nue_d - one;
         //doubflo ux_sq       = vx * vx;
         //doubflo uy_sq       = vy * vy;
         //doubflo uz_sq       = vz * vz;


         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         //D3Q7
         //doubflo ConcD       = f7ZERO+f7E+f7W+f7N+f7S+f7T+f7B;
         //(D7.f[0])[k  ] = f7ZERO*(one+omegaD)-omegaD*ConcD*(c1o3*(ae*(-three))-(ux_sq+uy_sq+uz_sq));
         //(D7.f[2])[kw ] = f7E   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)+vx*c1o2);
         //(D7.f[1])[k  ] = f7W   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)-vx*c1o2);
         //(D7.f[4])[ks ] = f7N   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)+vy*c1o2);
         //(D7.f[3])[k  ] = f7S   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)-vy*c1o2);
         //(D7.f[6])[kb ] = f7T   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)+vz*c1o2);
         //(D7.f[5])[k  ] = f7B   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)-vz*c1o2);
         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


         //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         ////D3Q27
         //doubflo ConcD   = (f27TNE+f27BSW)+(f27TSW+f27BNE)+(f27TSE+f27BNW)+(f27TNW+f27BSE)+
         //                  (f27NE+f27SW)+(f27NW+f27SE)+(f27TE+f27BW)+(f27BE+f27TW)+(f27TN+f27BS)+(f27BN+f27TS)+
         //                  (f27E+f27W)+(f27N+f27S)+(f27T+f27B)+f27ZERO;
         //doubflo cusq    =  c3o2*(vx1*vx1+vx2*vx2+vx3*vx3);

         //(D27.f[ dirE   ])[k   ] = f27W    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(-vx1        )+c9over2*(-vx1        )*(-vx1        )-cusq);                                                                     
         //(D27.f[ dirW   ])[kw  ] = f27E    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*( vx1        )+c9over2*( vx1        )*( vx1        )-cusq);                                                                     
         //(D27.f[ dirN   ])[k   ] = f27S    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(   -vx2     )+c9over2*(    -vx2    )*(    -vx2    )-cusq);
         //(D27.f[ dirS   ])[ks  ] = f27N    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(    vx2     )+c9over2*(     vx2    )*(     vx2    )-cusq);
         //(D27.f[ dirT   ])[k   ] = f27B    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(        -vx3)+c9over2*(        -vx3)*(        -vx3)-cusq);
         //(D27.f[ dirB   ])[kb  ] = f27T    *(one-omegaD)+omegaD* c2over27* ConcD*(one+three*(         vx3)+c9over2*(         vx3)*(         vx3)-cusq);
         //(D27.f[ dirNE  ])[k   ] = f27SW   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(-vx1-vx2    )+c9over2*(-vx1-vx2    )*(-vx1-vx2    )-cusq);
         //(D27.f[ dirSW  ])[ksw ] = f27NE   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*( vx1+vx2    )+c9over2*( vx1+vx2    )*( vx1+vx2    )-cusq);
         //(D27.f[ dirSE  ])[ks  ] = f27NW   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(-vx1+vx2    )+c9over2*(-vx1+vx2    )*(-vx1+vx2    )-cusq);
         //(D27.f[ dirNW  ])[kw  ] = f27SE   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*( vx1-vx2    )+c9over2*( vx1-vx2    )*( vx1-vx2    )-cusq);
         //(D27.f[ dirTE  ])[k   ] = f27BW   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(-vx1    -vx3)+c9over2*(-vx1    -vx3)*(-vx1    -vx3)-cusq);
         //(D27.f[ dirBW  ])[kbw ] = f27TE   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*( vx1    +vx3)+c9over2*( vx1    +vx3)*( vx1    +vx3)-cusq);
         //(D27.f[ dirBE  ])[kb  ] = f27TW   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(-vx1    +vx3)+c9over2*(-vx1    +vx3)*(-vx1    +vx3)-cusq);
         //(D27.f[ dirTW  ])[kw  ] = f27BE   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*( vx1    -vx3)+c9over2*( vx1    -vx3)*( vx1    -vx3)-cusq);
         //(D27.f[ dirTN  ])[k   ] = f27BS   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(    -vx2-vx3)+c9over2*(    -vx2-vx3)*(    -vx2-vx3)-cusq);
         //(D27.f[ dirBS  ])[kbs ] = f27TN   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(     vx2+vx3)+c9over2*(     vx2+vx3)*(     vx2+vx3)-cusq);
         //(D27.f[ dirBN  ])[kb  ] = f27TS   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(    -vx2+vx3)+c9over2*(    -vx2+vx3)*(    -vx2+vx3)-cusq);
         //(D27.f[ dirTS  ])[ks  ] = f27BN   *(one-omegaD)+omegaD* c1over54* ConcD*(one+three*(     vx2-vx3)+c9over2*(     vx2-vx3)*(     vx2-vx3)-cusq);
         //(D27.f[ dirZERO])[k   ] = f27ZERO *(one-omegaD)+omegaD* c8over27* ConcD*(one-cusq);
         //(D27.f[ dirTNE ])[k   ] = f27BSW  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*(-vx1-vx2-vx3)+c9over2*(-vx1-vx2-vx3)*(-vx1-vx2-vx3)-cusq);
         //(D27.f[ dirTSE ])[ks  ] = f27BNW  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*(-vx1+vx2-vx3)+c9over2*(-vx1+vx2-vx3)*(-vx1+vx2-vx3)-cusq);
         //(D27.f[ dirBNE ])[kb  ] = f27TSW  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*(-vx1-vx2+vx3)+c9over2*(-vx1-vx2+vx3)*(-vx1-vx2+vx3)-cusq);
         //(D27.f[ dirBSE ])[kbs ] = f27TNW  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*(-vx1+vx2+vx3)+c9over2*(-vx1+vx2+vx3)*(-vx1+vx2+vx3)-cusq);
         //(D27.f[ dirTNW ])[kw  ] = f27BSE  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*( vx1-vx2-vx3)+c9over2*( vx1-vx2-vx3)*( vx1-vx2-vx3)-cusq);
         //(D27.f[ dirTSW ])[ksw ] = f27BNE  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*( vx1+vx2-vx3)+c9over2*( vx1+vx2-vx3)*( vx1+vx2-vx3)-cusq);
         //(D27.f[ dirBNW ])[kbw ] = f27TSE  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*( vx1-vx2+vx3)+c9over2*( vx1-vx2+vx3)*( vx1-vx2+vx3)-cusq);
         //(D27.f[ dirBSW ])[kbsw] = f27TNE  *(one-omegaD)+omegaD* c1over216*ConcD*(one+three*( vx1+vx2+vx3)+c9over2*( vx1+vx2+vx3)*( vx1+vx2+vx3)-cusq);
         //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		 	doubflo oMdrho = zero;//one; // comp special
			doubflo m0, m1, m2;
			doubflo vx2=vvx*vvx;
			doubflo vy2=vvy*vvy;
			doubflo vz2=vvz*vvz;

			////////////////////////////////////////////////////////////////////////////////////
			//Hin
			////////////////////////////////////////////////////////////////////////////////////
			// mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m2    = mfaaa	+ mfaac;
			m1    = mfaac	- mfaaa;
			m0    = m2		+ mfaab;
			mfaaa = m0;
			m0   += c1o36 * oMdrho;	
			mfaab = m1 -		m0 * vvz;
			mfaac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfabc;
			m1    = mfabc  - mfaba;
			m0    = m2		+ mfabb;
			mfaba = m0;
			m0   += c1o9 * oMdrho;
			mfabb = m1 -		m0 * vvz;
			mfabc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfacc;
			m1    = mfacc  - mfaca;
			m0    = m2		+ mfacb;
			mfaca = m0;
			m0   += c1o36 * oMdrho;
			mfacb = m1 -		m0 * vvz;
			mfacc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbac;
			m1    = mfbac	- mfbaa;
			m0    = m2		+ mfbab;
			mfbaa = m0;
			m0   += c1o9 * oMdrho;
			mfbab = m1 -		m0 * vvz;
			mfbac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbba  + mfbbc;
			m1    = mfbbc  - mfbba;
			m0    = m2		+ mfbbb;
			mfbba = m0;
			m0   += c4o9 * oMdrho;
			mfbbb = m1 -		m0 * vvz;
			mfbbc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbca  + mfbcc;
			m1    = mfbcc  - mfbca;
			m0    = m2		+ mfbcb;
			mfbca = m0;
			m0   += c1o9 * oMdrho;
			mfbcb = m1 -		m0 * vvz;
			mfbcc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcac;
			m1    = mfcac	- mfcaa;
			m0    = m2		+ mfcab;
			mfcaa = m0;
			m0   += c1o36 * oMdrho;
			mfcab = m1 -		m0 * vvz;
			mfcac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcba  + mfcbc;
			m1    = mfcbc  - mfcba;
			m0    = m2		+ mfcbb;
			mfcba = m0;
			m0   += c1o9 * oMdrho;
			mfcbb = m1 -		m0 * vvz;
			mfcbc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcca  + mfccc;
			m1    = mfccc  - mfcca;
			m0    = m2		+ mfccb;
			mfcca = m0;
			m0   += c1o36 * oMdrho;
			mfccb = m1 -		m0 * vvz;
			mfccc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2    = mfaaa	+ mfaca;
			m1    = mfaca	- mfaaa;
			m0    = m2		+ mfaba;
			mfaaa = m0;
			m0   += c1o6 * oMdrho;
			mfaba = m1 -		m0 * vvy;
			mfaca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab  + mfacb;
			m1    = mfacb  - mfaab;
			m0    = m2		+ mfabb;
			mfaab = m0;
			mfabb = m1 -		m0 * vvy;
			mfacb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac  + mfacc;
			m1    = mfacc  - mfaac;
			m0    = m2		+ mfabc;
			mfaac = m0;
			m0   += c1o18 * oMdrho;
			mfabc = m1 -		m0 * vvy;
			mfacc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbaa	+ mfbca;
			m1    = mfbca	- mfbaa;
			m0    = m2		+ mfbba;
			mfbaa = m0;
			m0   += c2o3 * oMdrho;
			mfbba = m1 -		m0 * vvy;
			mfbca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbab  + mfbcb;
			m1    = mfbcb  - mfbab;
			m0    = m2		+ mfbbb;
			mfbab = m0;
			mfbbb = m1 -		m0 * vvy;
			mfbcb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfbac  + mfbcc;
			m1    = mfbcc  - mfbac;
			m0    = m2		+ mfbbc;
			mfbac = m0;
			m0   += c2o9 * oMdrho;
			mfbbc = m1 -		m0 * vvy;
			mfbcc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcaa	+ mfcca;
			m1    = mfcca	- mfcaa;
			m0    = m2		+ mfcba;
			mfcaa = m0;
			m0   += c1o6 * oMdrho;
			mfcba = m1 -		m0 * vvy;
			mfcca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcab  + mfccb;
			m1    = mfccb  - mfcab;
			m0    = m2		+ mfcbb;
			mfcab = m0;
			mfcbb = m1 -		m0 * vvy;
			mfccb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfcac  + mfccc;
			m1    = mfccc  - mfcac;
			m0    = m2		+ mfcbc;
			mfcac = m0;
			m0   += c1o18 * oMdrho;
			mfcbc = m1 -		m0 * vvy;
			mfccc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2    = mfaaa	+ mfcaa;
			m1    = mfcaa	- mfaaa;
			m0    = m2		+ mfbaa;
			mfaaa = m0;
			m0   += one* oMdrho;
			mfbaa = m1 -		m0 * vvx;
			mfcaa = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaba  + mfcba;
			m1    = mfcba  - mfaba;
			m0    = m2		+ mfbba;
			mfaba = m0;
			mfbba = m1 -		m0 * vvx;
			mfcba = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaca  + mfcca;
			m1    = mfcca  - mfaca;
			m0    = m2		+ mfbca;
			mfaca = m0;
			m0   += c1o3 * oMdrho;
			mfbca = m1 -		m0 * vvx;
			mfcca = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaab	+ mfcab;
			m1    = mfcab	- mfaab;
			m0    = m2		+ mfbab;
			mfaab = m0;
			mfbab = m1 -		m0 * vvx;
			mfcab = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabb  + mfcbb;
			m1    = mfcbb  - mfabb;
			m0    = m2		+ mfbbb;
			mfabb = m0;
			mfbbb = m1 -		m0 * vvx;
			mfcbb = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacb  + mfccb;
			m1    = mfccb  - mfacb;
			m0    = m2		+ mfbcb;
			mfacb = m0;
			mfbcb = m1 -		m0 * vvx;
			mfccb = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfaac	+ mfcac;
			m1    = mfcac	- mfaac;
			m0    = m2		+ mfbac;
			mfaac = m0;
			m0   += c1o3 * oMdrho;
			mfbac = m1 -		m0 * vvx;
			mfcac = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfabc  + mfcbc;
			m1    = mfcbc  - mfabc;
			m0    = m2		+ mfbbc;
			mfabc = m0;
			mfbbc = m1 -		m0 * vvx;
			mfcbc = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2    = mfacc  + mfccc;
			m1    = mfccc  - mfacc;
			m0    = m2		+ mfbcc;
			mfacc = m0;
			m0   += c1o9 * oMdrho;
			mfbcc = m1 -		m0 * vvx;
			mfccc = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////

			//if(mfaaa < zero) omegaD = one;
			doubflo limit=nine*omegaD*omegaD*(mfbaa*mfbaa+mfaba*mfaba+mfaab*mfaab);
			//doubflo CC=c1o2;
			//if ((two*mfaaa*mfaaa<limit)) omegaD=two / (six * (diffusivity+((limit/(1.0e-10f+two*mfaaa*mfaaa)-one)*(c1o6-diffusivity))*c1o2) + one);
			if ((two*mfaaa*mfaaa<limit)) omegaD=one;
			//omegaD = two / (six * (diffusivity+CC*limit) + one);

			//mfaaa = c1o2;
			//trans 3.
			doubflo Mabc=mfabc-mfaba*c1o3;
			doubflo Mbca=mfbca-mfbaa*c1o3;
			doubflo Macb=mfacb-mfaab*c1o3;
			doubflo Mcba=mfcba-mfaba*c1o3;
			doubflo Mcab=mfcab-mfaab*c1o3;
			doubflo Mbac=mfbac-mfbaa*c1o3;
			//trans 5.
			doubflo Mcbc=mfcbc-mfaba*c1o9;
			doubflo Mbcc=mfbcc-mfbaa*c1o9;
			doubflo Mccb=mfccb-mfaab*c1o9;

			//1.
			mfbaa *= one - omegaD;
			mfaba *= one - omegaD;
			mfaab *= one - omegaD;

			//3.
			//mfbca *= one - omegaD;
			//mfbac *= one - omegaD;
			//mfcba *= one - omegaD;
			//mfabc *= one - omegaD;
			//mfcab *= one - omegaD;
			//mfacb *= one - omegaD;

			//mfbbb *= one - omegaD; 
			Mabc  = zero;
			Mbca  = zero;
			Macb  = zero;
			Mcba  = zero;
			Mcab  = zero;
			Mbac  = zero;
			mfbbb = zero;

			//5.
			//mfbcc *= one - omegaD;
			//mfcbc *= one - omegaD;
			//mfccb *= one - omegaD;
			Mcbc = zero;
			Mbcc = zero;
			Mccb = zero;

			//2.
			mfbba = zero;
			mfbab = zero;
			mfabb = zero;

			mfcaa = c1o3 * drho;
			mfaca = c1o3 * drho;
			mfaac = c1o3 * drho;

			//4.
			mfacc = c1o9 * drho;
			mfcac = c1o9 * drho;
			mfcca = c1o9 * drho;

			mfcbb = zero;
			mfbcb = zero;
			mfbbc = zero;

			//6.
			mfccc = c1o27 * drho;

			//3.
			mfabc=Mabc+mfaba*c1o3;
			mfbca=Mbca+mfbaa*c1o3;
			mfacb=Macb+mfaab*c1o3;
			mfcba=Mcba+mfaba*c1o3;
			mfcab=Mcab+mfaab*c1o3;
			mfbac=Mbac+mfbaa*c1o3;
			//5.	  
			mfcbc=Mcbc+mfaba*c1o9;
			mfbcc=Mbcc+mfbaa*c1o9;
			mfccb=Mccb+mfaab*c1o9;

			////////////////////////////////////////////////////////////////////////////////////
			//back
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m0 =  mfaac * c1o2 +      mfaab * (vvz - c1o2) + (mfaaa + one* oMdrho) * (     vz2 - vvz) * c1o2; 
			m1 = -mfaac        - two* mfaab *  vvz         +  mfaaa                * (one- vz2)              - one* oMdrho * vz2; 
			m2 =  mfaac * c1o2 +      mfaab * (vvz + c1o2) + (mfaaa + one* oMdrho) * (     vz2 + vvz) * c1o2;
			mfaaa = m0;
			mfaab = m1;
			mfaac = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfabc * c1o2 +      mfabb * (vvz - c1o2) + mfaba * (     vz2 - vvz) * c1o2; 
			m1 = -mfabc        - two* mfabb *  vvz         + mfaba * (one- vz2); 
			m2 =  mfabc * c1o2 +      mfabb * (vvz + c1o2) + mfaba * (     vz2 + vvz) * c1o2;
			mfaba = m0;
			mfabb = m1;
			mfabc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfacc * c1o2 +      mfacb * (vvz - c1o2) + (mfaca + c1o3 * oMdrho) * (     vz2 - vvz) * c1o2; 
			m1 = -mfacc        - two* mfacb *  vvz         +  mfaca                  * (one- vz2)              - c1o3 * oMdrho * vz2; 
			m2 =  mfacc * c1o2 +      mfacb * (vvz + c1o2) + (mfaca + c1o3 * oMdrho) * (     vz2 + vvz) * c1o2;
			mfaca = m0;
			mfacb = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfbac * c1o2 +      mfbab * (vvz - c1o2) + mfbaa * (     vz2 - vvz) * c1o2; 
			m1 = -mfbac        - two* mfbab *  vvz         + mfbaa * (one- vz2); 
			m2 =  mfbac * c1o2 +      mfbab * (vvz + c1o2) + mfbaa * (     vz2 + vvz) * c1o2;
			mfbaa = m0;
			mfbab = m1;
			mfbac = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 =  mfbbc * c1o2 +      mfbbb * (vvz - c1o2) + mfbba * (     vz2 - vvz) * c1o2; 
			m1 = -mfbbc        - two* mfbbb *  vvz         + mfbba * (one- vz2); 
			m2 =  mfbbc * c1o2 +      mfbbb * (vvz + c1o2) + mfbba * (     vz2 + vvz) * c1o2;
			mfbba = m0;
			mfbbb = m1;
			mfbbc = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 =  mfbcc * c1o2 +      mfbcb * (vvz - c1o2) + mfbca * (     vz2 - vvz) * c1o2; 
			m1 = -mfbcc        - two* mfbcb *  vvz         + mfbca * (one- vz2); 
			m2 =  mfbcc * c1o2 +      mfbcb * (vvz + c1o2) + mfbca * (     vz2 + vvz) * c1o2;
			mfbca = m0;
			mfbcb = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcac * c1o2 +      mfcab * (vvz - c1o2) + (mfcaa + c1o3 * oMdrho) * (     vz2 - vvz) * c1o2; 
			m1 = -mfcac        - two* mfcab *  vvz         +  mfcaa                  * (one- vz2)              - c1o3 * oMdrho * vz2; 
			m2 =  mfcac * c1o2 +      mfcab * (vvz + c1o2) + (mfcaa + c1o3 * oMdrho) * (     vz2 + vvz) * c1o2;
			mfcaa = m0;
			mfcab = m1;
			mfcac = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 =  mfcbc * c1o2 +      mfcbb * (vvz - c1o2) + mfcba * (     vz2 - vvz) * c1o2; 
			m1 = -mfcbc        - two* mfcbb *  vvz         + mfcba * (one- vz2); 
			m2 =  mfcbc * c1o2 +      mfcbb * (vvz + c1o2) + mfcba * (     vz2 + vvz) * c1o2;
			mfcba = m0;
			mfcbb = m1;
			mfcbc = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 =  mfccc * c1o2 +      mfccb * (vvz - c1o2) + (mfcca + c1o9 * oMdrho) * (     vz2 - vvz) * c1o2; 
			m1 = -mfccc        - two* mfccb *  vvz         +  mfcca                  * (one- vz2)              - c1o9 * oMdrho * vz2; 
			m2 =  mfccc * c1o2 +      mfccb * (vvz + c1o2) + (mfcca + c1o9 * oMdrho) * (     vz2 + vvz) * c1o2;
			mfcca = m0;
			mfccb = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m0 =  mfaca * c1o2 +      mfaba * (vvy - c1o2) + (mfaaa + c1o6 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfaca        - two* mfaba *  vvy         +  mfaaa                  * (one- vy2)              - c1o6 * oMdrho * vy2; 
			m2 =  mfaca * c1o2 +      mfaba * (vvy + c1o2) + (mfaaa + c1o6 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfaaa = m0;
			mfaba = m1;
			mfaca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfacb * c1o2 +      mfabb * (vvy - c1o2) + (mfaab + c2o3 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfacb        - two* mfabb *  vvy         +  mfaab                  * (one- vy2)              - c2o3 * oMdrho * vy2; 
			m2 =  mfacb * c1o2 +      mfabb * (vvy + c1o2) + (mfaab + c2o3 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfaab = m0;
			mfabb = m1;
			mfacb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfacc * c1o2 +      mfabc * (vvy - c1o2) + (mfaac + c1o6 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfacc        - two* mfabc *  vvy         +  mfaac                  * (one- vy2)              - c1o6 * oMdrho * vy2; 
			m2 =  mfacc * c1o2 +      mfabc * (vvy + c1o2) + (mfaac + c1o6 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfaac = m0;
			mfabc = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfbca * c1o2 +      mfbba * (vvy - c1o2) + mfbaa * (     vy2 - vvy) * c1o2; 
			m1 = -mfbca        - two* mfbba *  vvy         + mfbaa * (one- vy2); 
			m2 =  mfbca * c1o2 +      mfbba * (vvy + c1o2) + mfbaa * (     vy2 + vvy) * c1o2;
			mfbaa = m0;
			mfbba = m1;
			mfbca = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 =  mfbcb * c1o2 +      mfbbb * (vvy - c1o2) + mfbab * (     vy2 - vvy) * c1o2; 
			m1 = -mfbcb        - two* mfbbb *  vvy         + mfbab * (one- vy2); 
			m2 =  mfbcb * c1o2 +      mfbbb * (vvy + c1o2) + mfbab * (     vy2 + vvy) * c1o2;
			mfbab = m0;
			mfbbb = m1;
			mfbcb = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 =  mfbcc * c1o2 +      mfbbc * (vvy - c1o2) + mfbac * (     vy2 - vvy) * c1o2; 
			m1 = -mfbcc        - two* mfbbc *  vvy         + mfbac * (one- vy2); 
			m2 =  mfbcc * c1o2 +      mfbbc * (vvy + c1o2) + mfbac * (     vy2 + vvy) * c1o2;
			mfbac = m0;
			mfbbc = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcca * c1o2 +      mfcba * (vvy - c1o2) + (mfcaa + c1o18 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfcca        - two* mfcba *  vvy         +  mfcaa                   * (one- vy2)              - c1o18 * oMdrho * vy2; 
			m2 =  mfcca * c1o2 +      mfcba * (vvy + c1o2) + (mfcaa + c1o18 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfcaa = m0;
			mfcba = m1;
			mfcca = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 =  mfccb * c1o2 +      mfcbb * (vvy - c1o2) + (mfcab + c2o9 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfccb        - two* mfcbb *  vvy         +  mfcab                  * (one- vy2)              - c2o9 * oMdrho * vy2; 
			m2 =  mfccb * c1o2 +      mfcbb * (vvy + c1o2) + (mfcab + c2o9 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfcab = m0;
			mfcbb = m1;
			mfccb = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 =  mfccc * c1o2 +      mfcbc * (vvy - c1o2) + (mfcac + c1o18 * oMdrho) * (     vy2 - vvy) * c1o2; 
			m1 = -mfccc        - two* mfcbc *  vvy         +  mfcac                   * (one- vy2)              - c1o18 * oMdrho * vy2; 
			m2 =  mfccc * c1o2 +      mfcbc * (vvy + c1o2) + (mfcac + c1o18 * oMdrho) * (     vy2 + vvy) * c1o2;
			mfcac = m0;
			mfcbc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m0 =  mfcaa * c1o2 +      mfbaa * (vvx - c1o2) + (mfaaa + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcaa        - two* mfbaa *  vvx         +  mfaaa                   * (one- vx2)              - c1o36 * oMdrho * vx2; 
			m2 =  mfcaa * c1o2 +      mfbaa * (vvx + c1o2) + (mfaaa + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaaa = m0;
			mfbaa = m1;
			mfcaa = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcba * c1o2 +      mfbba * (vvx - c1o2) + (mfaba + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcba        - two* mfbba *  vvx         +  mfaba                  * (one- vx2)              - c1o9 * oMdrho * vx2; 
			m2 =  mfcba * c1o2 +      mfbba * (vvx + c1o2) + (mfaba + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaba = m0;
			mfbba = m1;
			mfcba = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcca * c1o2 +      mfbca * (vvx - c1o2) + (mfaca + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcca        - two* mfbca *  vvx         +  mfaca                   * (one- vx2)              - c1o36 * oMdrho * vx2; 
			m2 =  mfcca * c1o2 +      mfbca * (vvx + c1o2) + (mfaca + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaca = m0;
			mfbca = m1;
			mfcca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcab * c1o2 +      mfbab * (vvx - c1o2) + (mfaab + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcab        - two* mfbab *  vvx         +  mfaab                  * (one- vx2)              - c1o9 * oMdrho * vx2; 
			m2 =  mfcab * c1o2 +      mfbab * (vvx + c1o2) + (mfaab + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaab = m0;
			mfbab = m1;
			mfcab = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 =  mfcbb * c1o2 +      mfbbb * (vvx - c1o2) + (mfabb + c4o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcbb        - two* mfbbb *  vvx         +  mfabb                  * (one- vx2)              - c4o9 * oMdrho * vx2; 
			m2 =  mfcbb * c1o2 +      mfbbb * (vvx + c1o2) + (mfabb + c4o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfabb = m0;
			mfbbb = m1;
			mfcbb = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 =  mfccb * c1o2 +      mfbcb * (vvx - c1o2) + (mfacb + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfccb        - two* mfbcb *  vvx         +  mfacb                  * (one- vx2)              - c1o9 * oMdrho * vx2; 
			m2 =  mfccb * c1o2 +      mfbcb * (vvx + c1o2) + (mfacb + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfacb = m0;
			mfbcb = m1;
			mfccb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 =  mfcac * c1o2 +      mfbac * (vvx - c1o2) + (mfaac + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcac        - two* mfbac *  vvx         +  mfaac                   * (one- vx2)              - c1o36 * oMdrho * vx2; 
			m2 =  mfcac * c1o2 +      mfbac * (vvx + c1o2) + (mfaac + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfaac = m0;
			mfbac = m1;
			mfcac = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 =  mfcbc * c1o2 +      mfbbc * (vvx - c1o2) + (mfabc + c1o9 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfcbc        - two* mfbbc *  vvx         +  mfabc                  * (one- vx2)              - c1o9 * oMdrho * vx2; 
			m2 =  mfcbc * c1o2 +      mfbbc * (vvx + c1o2) + (mfabc + c1o9 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfabc = m0;
			mfbbc = m1;
			mfcbc = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 =  mfccc * c1o2 +      mfbcc * (vvx - c1o2) + (mfacc + c1o36 * oMdrho) * (     vx2 - vvx) * c1o2; 
			m1 = -mfccc        - two* mfbcc *  vvx         +  mfacc                   * (one- vx2)              - c1o36 * oMdrho * vx2; 
			m2 =  mfccc * c1o2 +      mfbcc * (vvx + c1o2) + (mfacc + c1o36 * oMdrho) * (     vx2 + vvx) * c1o2;
			mfacc = m0;
			mfbcc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			(D27.f[ dirE   ])[k   ] = mfabb;                                                                   
			(D27.f[ dirW   ])[kw  ] = mfcbb;                                                                 
			(D27.f[ dirN   ])[k   ] = mfbab;
			(D27.f[ dirS   ])[ks  ] = mfbcb;
			(D27.f[ dirT   ])[k   ] = mfbba;
			(D27.f[ dirB   ])[kb  ] = mfbbc;
			(D27.f[ dirNE  ])[k   ] = mfaab;
			(D27.f[ dirSW  ])[ksw ] = mfccb;
			(D27.f[ dirSE  ])[ks  ] = mfacb;
			(D27.f[ dirNW  ])[kw  ] = mfcab;
			(D27.f[ dirTE  ])[k   ] = mfaba;
			(D27.f[ dirBW  ])[kbw ] = mfcbc;
			(D27.f[ dirBE  ])[kb  ] = mfabc;
			(D27.f[ dirTW  ])[kw  ] = mfcba;
			(D27.f[ dirTN  ])[k   ] = mfbaa;
			(D27.f[ dirBS  ])[kbs ] = mfbcc;
			(D27.f[ dirBN  ])[kb  ] = mfbac;
			(D27.f[ dirTS  ])[ks  ] = mfbca;
			(D27.f[ dirZERO])[k   ] = mfbbb;
			(D27.f[ dirTNE ])[k   ] = mfaaa;
			(D27.f[ dirTSE ])[ks  ] = mfaca;
			(D27.f[ dirBNE ])[kb  ] = mfaac;
			(D27.f[ dirBSE ])[kbs ] = mfacc;
			(D27.f[ dirTNW ])[kw  ] = mfcaa;
			(D27.f[ dirTSW ])[ksw ] = mfcca;
			(D27.f[ dirBNW ])[kbw ] = mfcac;
			(D27.f[ dirBSW ])[kbsw] = mfccc;
			////////////////////////////////////////////////////////////////////////////////////

      }                                                                                                                    
   }
}


























////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void LB_Kernel_AD_Incomp_7(  doubflo diffusivity,
												   unsigned int* bcMatD,
												   unsigned int* neighborX,
												   unsigned int* neighborY,
												   unsigned int* neighborZ,
												   doubflo* DDStart,
												   doubflo* DD7,
												   int size_Mat,
												   bool EvenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<size_Mat)
   {
      ////////////////////////////////////////////////////////////////////////////////
      unsigned int BC;
      BC        =   bcMatD[k];

      if( (BC != GEO_SOLID) && (BC != GEO_VOID))
      {
         Distributions27 D;
         if (EvenOrOdd==true)
         {
            D.f[dirE   ] = &DDStart[dirE   *size_Mat];
            D.f[dirW   ] = &DDStart[dirW   *size_Mat];
            D.f[dirN   ] = &DDStart[dirN   *size_Mat];
            D.f[dirS   ] = &DDStart[dirS   *size_Mat];
            D.f[dirT   ] = &DDStart[dirT   *size_Mat];
            D.f[dirB   ] = &DDStart[dirB   *size_Mat];
            D.f[dirNE  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirSW  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirTNE ] = &DDStart[dirTNE *size_Mat];
            D.f[dirTSW ] = &DDStart[dirTSW *size_Mat];
            D.f[dirTSE ] = &DDStart[dirTSE *size_Mat];
            D.f[dirTNW ] = &DDStart[dirTNW *size_Mat];
            D.f[dirBNE ] = &DDStart[dirBNE *size_Mat];
            D.f[dirBSW ] = &DDStart[dirBSW *size_Mat];
            D.f[dirBSE ] = &DDStart[dirBSE *size_Mat];
            D.f[dirBNW ] = &DDStart[dirBNW *size_Mat];
         }
         else
         {
            D.f[dirW   ] = &DDStart[dirE   *size_Mat];
            D.f[dirE   ] = &DDStart[dirW   *size_Mat];
            D.f[dirS   ] = &DDStart[dirN   *size_Mat];
            D.f[dirN   ] = &DDStart[dirS   *size_Mat];
            D.f[dirB   ] = &DDStart[dirT   *size_Mat];
            D.f[dirT   ] = &DDStart[dirB   *size_Mat];
            D.f[dirSW  ] = &DDStart[dirNE  *size_Mat];
            D.f[dirNE  ] = &DDStart[dirSW  *size_Mat];
            D.f[dirNW  ] = &DDStart[dirSE  *size_Mat];
            D.f[dirSE  ] = &DDStart[dirNW  *size_Mat];
            D.f[dirBW  ] = &DDStart[dirTE  *size_Mat];
            D.f[dirTE  ] = &DDStart[dirBW  *size_Mat];
            D.f[dirTW  ] = &DDStart[dirBE  *size_Mat];
            D.f[dirBE  ] = &DDStart[dirTW  *size_Mat];
            D.f[dirBS  ] = &DDStart[dirTN  *size_Mat];
            D.f[dirTN  ] = &DDStart[dirBS  *size_Mat];
            D.f[dirTS  ] = &DDStart[dirBN  *size_Mat];
            D.f[dirBN  ] = &DDStart[dirTS  *size_Mat];
            D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
            D.f[dirBSW ] = &DDStart[dirTNE *size_Mat];
            D.f[dirBNE ] = &DDStart[dirTSW *size_Mat];
            D.f[dirBNW ] = &DDStart[dirTSE *size_Mat];
            D.f[dirBSE ] = &DDStart[dirTNW *size_Mat];
            D.f[dirTSW ] = &DDStart[dirBNE *size_Mat];
            D.f[dirTNE ] = &DDStart[dirBSW *size_Mat];
            D.f[dirTNW ] = &DDStart[dirBSE *size_Mat];
            D.f[dirTSE ] = &DDStart[dirBNW *size_Mat];
         }

         Distributions7 D7;
         if (EvenOrOdd==true)
         {
            D7.f[0] = &DD7[0*size_Mat];
            D7.f[1] = &DD7[1*size_Mat];
            D7.f[2] = &DD7[2*size_Mat];
            D7.f[3] = &DD7[3*size_Mat];
            D7.f[4] = &DD7[4*size_Mat];
            D7.f[5] = &DD7[5*size_Mat];
            D7.f[6] = &DD7[6*size_Mat];
         }
         else
         {
            D7.f[0] = &DD7[0*size_Mat];
            D7.f[2] = &DD7[1*size_Mat];
            D7.f[1] = &DD7[2*size_Mat];
            D7.f[4] = &DD7[3*size_Mat];
            D7.f[3] = &DD7[4*size_Mat];
            D7.f[6] = &DD7[5*size_Mat];
            D7.f[5] = &DD7[6*size_Mat];
         }

         ////////////////////////////////////////////////////////////////////////////////
         //index
         unsigned int kw   = neighborX[k];
         unsigned int ks   = neighborY[k];
         unsigned int kb   = neighborZ[k];
         unsigned int ksw  = neighborY[kw];
         unsigned int kbw  = neighborZ[kw];
         unsigned int kbs  = neighborZ[ks];
         unsigned int kbsw = neighborZ[ksw];
         //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
         doubflo fW    =  (D.f[dirE   ])[k  ];//ke
         doubflo fE    =  (D.f[dirW   ])[kw ];
         doubflo fS    =  (D.f[dirN   ])[k  ];//kn
         doubflo fN    =  (D.f[dirS   ])[ks ];
         doubflo fB    =  (D.f[dirT   ])[k  ];//kt
         doubflo fT    =  (D.f[dirB   ])[kb ];
         doubflo fSW   =  (D.f[dirNE  ])[k  ];//kne
         doubflo fNE   =  (D.f[dirSW  ])[ksw];
         doubflo fNW   =  (D.f[dirSE  ])[ks ];//kse
         doubflo fSE   =  (D.f[dirNW  ])[kw ];//knw
         doubflo fBW   =  (D.f[dirTE  ])[k  ];//kte
         doubflo fTE   =  (D.f[dirBW  ])[kbw];
         doubflo fTW   =  (D.f[dirBE  ])[kb ];//kbe
         doubflo fBE   =  (D.f[dirTW  ])[kw ];//ktw
         doubflo fBS   =  (D.f[dirTN  ])[k  ];//ktn
         doubflo fTN   =  (D.f[dirBS  ])[kbs];
         doubflo fTS   =  (D.f[dirBN  ])[kb ];//kbn
         doubflo fBN   =  (D.f[dirTS  ])[ks ];//kts
         doubflo fZERO =  (D.f[dirZERO])[k  ];//kzero
         doubflo fBSW   = (D.f[dirTNE ])[k  ];//ktne
         doubflo fBNE   = (D.f[dirTSW ])[ksw];//ktsw
         doubflo fBNW   = (D.f[dirTSE ])[ks ];//ktse
         doubflo fBSE   = (D.f[dirTNW ])[kw ];//ktnw
         doubflo fTSW   = (D.f[dirBNE ])[kb ];//kbne
         doubflo fTNE   = (D.f[dirBSW ])[kbsw];
         doubflo fTNW   = (D.f[dirBSE ])[kbs];//kbse
         doubflo fTSE   = (D.f[dirBNW ])[kbw];//kbnw
         //doubflo fE    =  (D.f[dirE   ])[k  ];//ke
         //doubflo fW    =  (D.f[dirW   ])[kw ];
         //doubflo fN    =  (D.f[dirN   ])[k  ];//kn
         //doubflo fS    =  (D.f[dirS   ])[ks ];
         //doubflo fT    =  (D.f[dirT   ])[k  ];//kt
         //doubflo fB    =  (D.f[dirB   ])[kb ];
         //doubflo fNE   =  (D.f[dirNE  ])[k  ];//kne
         //doubflo fSW   =  (D.f[dirSW  ])[ksw];
         //doubflo fSE   =  (D.f[dirSE  ])[ks ];//kse
         //doubflo fNW   =  (D.f[dirNW  ])[kw ];//knw
         //doubflo fTE   =  (D.f[dirTE  ])[k  ];//kte
         //doubflo fBW   =  (D.f[dirBW  ])[kbw];
         //doubflo fBE   =  (D.f[dirBE  ])[kb ];//kbe
         //doubflo fTW   =  (D.f[dirTW  ])[kw ];//ktw
         //doubflo fTN   =  (D.f[dirTN  ])[k  ];//ktn
         //doubflo fBS   =  (D.f[dirBS  ])[kbs];
         //doubflo fBN   =  (D.f[dirBN  ])[kb ];//kbn
         //doubflo fTS   =  (D.f[dirTS  ])[ks ];//kts
         //doubflo fZERO =  (D.f[dirZERO])[k  ];//kzero
         //doubflo fTNE   = (D.f[dirTNE ])[k  ];//ktne
         //doubflo fTSW   = (D.f[dirTSW ])[ksw];//ktsw
         //doubflo fTSE   = (D.f[dirTSE ])[ks ];//ktse
         //doubflo fTNW   = (D.f[dirTNW ])[kw ];//ktnw
         //doubflo fBNE   = (D.f[dirBNE ])[kb ];//kbne
         //doubflo fBSW   = (D.f[dirBSW ])[kbsw];
         //doubflo fBSE   = (D.f[dirBSE ])[kbs];//kbse
         //doubflo fBNW   = (D.f[dirBNW ])[kbw];//kbnw
         ////////////////////////////////////////////////////////////////////////////////
         doubflo f7ZERO =  (D7.f[0])[k  ];
         doubflo f7E    =  (D7.f[1])[k  ];
         doubflo f7W    =  (D7.f[2])[kw ];
         doubflo f7N    =  (D7.f[3])[k  ];
         doubflo f7S    =  (D7.f[4])[ks ];
         doubflo f7T    =  (D7.f[5])[k  ];
         doubflo f7B    =  (D7.f[6])[kb ];
         ////////////////////////////////////////////////////////////////////////////////
         doubflo vx     =  ((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
         doubflo vy     =  ((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
         doubflo vz     =  ((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
		 ////d�rrrrrty !!!!!!!!!!!!!
   //      doubflo vx     =  ten * ((fTNE-fBSW)+(fBNE-fTSW)+(fTSE-fBNW)+(fBSE-fTNW) +(fNE-fSW)+(fSE-fNW)+(fTE-fBW)+(fBE-fTW)+(fE-fW));
   //      doubflo vy     =  ten * ((fTNE-fBSW)+(fBNE-fTSW)+(fBNW-fTSE)+(fTNW-fBSE) +(fNE-fSW)+(fNW-fSE)+(fTN-fBS)+(fBN-fTS)+(fN-fS));
   //      doubflo vz     =  ten * ((fTNE-fBSW)+(fTSW-fBNE)+(fTSE-fBNW)+(fTNW-fBSE) +(fTE-fBW)+(fTW-fBE)+(fTN-fBS)+(fTS-fBN)+(fT-fB));
         ////////////////////////////////////////////////////////////////////////////////
         doubflo ux_sq       = vx * vx;
         doubflo uy_sq       = vy * vy;
         doubflo uz_sq       = vz * vz;
         ////////////////////////////////////////////////////////////////////////////////
		 //BGK
         //doubflo omegaD     = -three + sqrt(three); !!!!!!!!!!!!!!Achtung!!!!!!!!!!!!!!!!!! anderes Vorzeichen als in den Randbedingungen
         //doubflo Lam         = -(c1o2+one/omegaD);
         //doubflo nue_d       = Lam/three;
         //doubflo ae          = diffusivity/nue_d - one;

         //doubflo ConcD       = f7ZERO+f7E+f7W+f7N+f7S+f7T+f7B;

         //(D7.f[0])[k  ] = f7ZERO*(one+omegaD)-omegaD*ConcD*(c1o3*(ae*(-three))-(ux_sq+uy_sq+uz_sq));
         //(D7.f[2])[kw ] = f7E   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)+vx*c1o2);
         //(D7.f[1])[k  ] = f7W   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(ux_sq)-vx*c1o2);
         //(D7.f[4])[ks ] = f7N   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)+vy*c1o2);
         //(D7.f[3])[k  ] = f7S   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uy_sq)-vy*c1o2);
         //(D7.f[6])[kb ] = f7T   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)+vz*c1o2);
         //(D7.f[5])[k  ] = f7B   *(one+omegaD)-omegaD*ConcD*(c1o6*(ae+one)+c1o2*(uz_sq)-vz*c1o2);

         ////////////////////////////////////////////////////////////////////////////////
		 //TRT  Yoshida Kernel - based on Ying
         doubflo cs2         = c1o4;
         doubflo Lam         = diffusivity*four;//diffusivity/(one)/cs2;
         doubflo omegaD      = - one / (Lam + c1o2);
         doubflo ae          = zero;
         ////////////////////////////////////////////////////////////////////////////////
         doubflo ConcD       = f7ZERO+f7E+f7W+f7N+f7S+f7T+f7B;

		 doubflo Mom000 = f7ZERO + f7W + f7E + f7N + f7S + f7T + f7B; //1
         doubflo Mom100 = f7E - f7W;
         doubflo Mom010 = f7N - f7S;
         doubflo Mom001 = f7T - f7B;
         doubflo Mom222 = six*f7ZERO - f7W - f7E - f7N - f7S - f7T - f7B;
         doubflo Mom200 = two*f7W + two*f7E - f7N - f7S - f7T - f7B;
         doubflo Mom022 = f7N + f7S - f7T - f7B;

         doubflo Meq000 = ConcD;
         doubflo Meq100 = ConcD*vx;
         doubflo Meq010 = ConcD*vy;
         doubflo Meq001 = ConcD*vz;
         doubflo Meq222 = c3o4*ConcD;
         doubflo Meq200 = zero;
         doubflo Meq022 = zero;

         // relaxation TRT Yoshida

         // odd 
         Mom100 = omegaD * (Mom100-Meq100);
         Mom010 = omegaD * (Mom010-Meq010);
         Mom001 = omegaD * (Mom001-Meq001);
         
         // even
         Mom000 = -one*(Mom000-Meq000);
         Mom222 = -one*(Mom222-Meq222);
         Mom200 = -one*(Mom200-Meq200);
         Mom022 = -one*(Mom022-Meq022);
         
         //Back transformation to distributions
         f7ZERO = f7ZERO + c1o7*Mom000 + c1o7*Mom222;                                                  //1
         f7E    = f7E    + c1o7*Mom000 + c1o2*Mom100 - c1o6*c1o7*Mom222 + c1o6*Mom200;                 //2
         f7W    = f7W    + c1o7*Mom000 - c1o2*Mom100 - c1o6*c1o7*Mom222 + c1o6*Mom200;                 //3
         f7N    = f7N    + c1o7*Mom000 + c1o2*Mom010 - c1o6*c1o7*Mom222 - c1o12*Mom200 + c1o4 *Mom022; //4
         f7S    = f7S    + c1o7*Mom000 - c1o2*Mom010 - c1o6*c1o7*Mom222 - c1o12*Mom200 + c1o4 *Mom022; //5
         f7T    = f7T    + c1o7*Mom000 + c1o2*Mom001 - c1o6*c1o7*Mom222 - c1o12*Mom200 - c1o4 *Mom022; //6
         f7B    = f7B    + c1o7*Mom000 - c1o2*Mom001 - c1o6*c1o7*Mom222 - c1o12*Mom200 - c1o4 *Mom022; //7

         (D7.f[0])[k  ] = f7ZERO;
         (D7.f[2])[kw ] = f7E   ;
         (D7.f[1])[k  ] = f7W   ;
         (D7.f[4])[ks ] = f7N   ;
         (D7.f[3])[k  ] = f7S   ;
         (D7.f[6])[kb ] = f7T   ;
         (D7.f[5])[k  ] = f7B   ;
	  }                                                                                                                    
   }
}
////////////////////////////////////////////////////////////////////////////////


