#include "hip/hip_runtime.h"
#include "EnstrophyAnalyzer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cmath>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#include <iomanip>

#include "Core/Logger/Logger.h"

#include "Parameter\Parameter.h"
// includes, kernels
#include "GPU/GPU_Kernels.cuh"
#include "GPU/constant.h"

__global__                 void enstrophyKernel  ( real* veloX, real* veloY, real* veloZ, real* rho, uint* neighborX, uint* neighborY, uint* neighborZ, uint* neighborWSB, uint* geo, real* enstrophy, uint size_Mat );

__host__ __device__ inline void enstrophyFunction( real* veloX, real* veloY, real* veloZ, real* rho, uint* neighborX, uint* neighborY, uint* neighborZ, uint* neighborWSB, real* enstrophy, uint index );

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

bool EnstrophyAnalyzer::run(uint iter)
{
    if( iter % this->analyzeIter != 0 ) return false;

	int lev = 0;
	int size_Mat = this->para->getParD(lev)->size_Mat_SP;
	
	thrust::device_vector<real> enstrophy( size_Mat );

	unsigned int numberOfThreads = 128;
    int Grid = (size_Mat / numberOfThreads)+1;
    int Grid1, Grid2;
    if (Grid>512)
    {
       Grid1 = 512;
       Grid2 = (Grid/Grid1)+1;
    } 
    else
    {
       Grid1 = 1;
       Grid2 = Grid;
    }
    dim3 grid(Grid1, Grid2);
    dim3 threads(numberOfThreads, 1, 1 );

	printf("Enstrophy - Before LBCalcMacCompSP27\n");
     LBCalcMacCompSP27<<< grid, threads >>> (para->getParD(lev)->vx_SP,
											 para->getParD(lev)->vy_SP,
											 para->getParD(lev)->vz_SP,
											 para->getParD(lev)->rho_SP,
											 para->getParD(lev)->press_SP,
											 para->getParD(lev)->geoSP,
											 para->getParD(lev)->neighborX_SP,
											 para->getParD(lev)->neighborY_SP,
											 para->getParD(lev)->neighborZ_SP,
											 para->getParD(lev)->size_Mat_SP,
											 para->getParD(lev)->d0SP.f[0],
											 para->getParD(lev)->evenOrOdd); 
	 //hipDeviceSynchronize();
	 getLastCudaError("LBCalcMacSP27 execution failed"); 

	 printf("Enstrophy - Before enstrophyKernel\n");
	 enstrophyKernel <<< grid, threads >>> (para->getParD(lev)->vx_SP,
											para->getParD(lev)->vy_SP, 
											para->getParD(lev)->vz_SP, 
											para->getParD(lev)->rho_SP, 
											para->getParD(lev)->neighborX_SP,
											para->getParD(lev)->neighborY_SP,
											para->getParD(lev)->neighborZ_SP,
											para->getParD(lev)->neighborWSB_SP,
											para->getParD(lev)->geoSP,
											enstrophy.data().get(), 
											size_Mat);
	 hipDeviceSynchronize(); 
	 getLastCudaError("enstrophyKernel execution failed");

	printf("Enstrophy - before reduce\n");
	real EnstrophyTmp = thrust::reduce(enstrophy.begin(), enstrophy.end(), zero, thrust::plus<real>());// / real(dataBase->perLevelCount[0].numberOfBulkCells);
	printf("Enstrophy - after reduce\n");

	this->enstrophyTimeSeries.push_back( EnstrophyTmp );
	printf("Enstrophy - after push_back\n");

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void enstrophyKernel(real* veloX, real* veloY, real* veloZ, real* rho, uint* neighborX, uint* neighborY, uint* neighborZ, uint* neighborWSB, uint* geo, real* enstrophy, uint size_Mat)
{
	////////////////////////////////////////////////////////////////////////////////
	uint index;                // Zugriff auf arrays im device
							   //
	uint tx = threadIdx.x;     // Thread index = lokaler i index
	uint by = blockIdx.x;      // Block index x
	uint bz = blockIdx.y;      // Block index y
	uint  x = tx + STARTOFFX;  // Globaler x-Index 
	uint  y = by + STARTOFFY;  // Globaler y-Index 
	uint  z = bz + STARTOFFZ;  // Globaler z-Index 

	const unsigned sizeX = blockDim.x;
	const unsigned sizeY = gridDim.x;
	const unsigned nx = sizeX + 2 * STARTOFFX;
	const unsigned ny = sizeY + 2 * STARTOFFY;

	index = nx*(ny*z + y) + x;
	////////////////////////////////////////////////////////////////////////////////
	enstrophy[index] = zero;
    if( index >= size_Mat) return;

	unsigned int BC;
	BC = geo[index];
	if (BC != GEO_FLUID) return;

    enstrophyFunction( veloX, veloY, veloZ, rho, neighborX, neighborY, neighborZ, neighborWSB, enstrophy, index );
}

__host__ __device__ void enstrophyFunction(real* veloX, real* veloY, real* veloZ, real* rho, uint* neighborX, uint* neighborY, uint* neighborZ, uint* neighborWSB, real* enstrophy, uint index)
{
	//////////////////////////////////////////////////////////////////////////////
	//neighbor index
	uint k = index;
	uint kPx = neighborX[k];
	uint kPy = neighborY[k];
	uint kPz = neighborZ[k];
	uint kMxyz = neighborWSB[k];
	uint kMx = neighborZ[neighborY[kMxyz]];
	uint kMy = neighborZ[neighborX[kMxyz]];
	uint kMz = neighborY[neighborX[kMxyz]];
	//getVeloX//
	real veloXNeighborPx = veloX[kPx];
	real veloXNeighborMx = veloX[kMx];
	real veloXNeighborPy = veloX[kPy];
	real veloXNeighborMy = veloX[kMy];
	real veloXNeighborPz = veloX[kPz];
	real veloXNeighborMz = veloX[kMz];
	//getVeloY//
	real veloYNeighborPx = veloY[kPx];
	real veloYNeighborMx = veloY[kMx];
	real veloYNeighborPy = veloY[kPy];
	real veloYNeighborMy = veloY[kMy];
	real veloYNeighborPz = veloY[kPz];
	real veloYNeighborMz = veloY[kMz];
	//getVeloZ//
	real veloZNeighborPx = veloZ[kPx];
	real veloZNeighborMx = veloZ[kMx];
	real veloZNeighborPy = veloZ[kPy];
	real veloZNeighborMy = veloZ[kMy];
	real veloZNeighborPz = veloZ[kPz];
	real veloZNeighborMz = veloZ[kMz];
	//getVeloLocal//
	real veloLocalX = veloX[k];
	real veloLocalY = veloY[k];
	real veloLocalZ = veloZ[k];
	//////////////////////////////////////////////////////////////////////////////
	real dxvx = zero;
	real dyvx = zero;
	real dzvx = zero;
	real dxvy = zero;
	real dyvy = zero;
	real dzvy = zero;
	real dxvz = zero;
	real dyvz = zero;
	real dzvz = zero;

	dxvy = (veloYNeighborPx - veloYNeighborMx) / two;
	dxvz = (veloZNeighborPx - veloZNeighborMx) / two;

	dyvx = (veloXNeighborPy - veloXNeighborMy) / two;
	dyvz = (veloZNeighborPy - veloZNeighborMy) / two;

	dzvx = (veloXNeighborPz - veloXNeighborMz) / two;
	dzvy = (veloYNeighborPz - veloYNeighborMz) / two;

	real tmpX = dyvz - dzvy;
	real tmpY = dzvx - dxvz;
	real tmpZ = dxvy - dyvx;
    //////////////////////////////////////////////////////////////////////////

    enstrophy[ index ] = c1o2 * (rho[index] + one) * ( tmpX*tmpX + tmpY*tmpY + tmpZ*tmpZ );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

EnstrophyAnalyzer::EnstrophyAnalyzer(SPtr<Parameter> para, uint analyzeIter)
{
	this->para = para;
	this->analyzeIter = analyzeIter;
}

void EnstrophyAnalyzer::writeToFile( std::string filename )
{
    *logging::out << logging::Logger::INFO_INTERMEDIATE << "EnstrophyAnalyzer::writeToFile( " << filename << " )" << "\n";

    std::ofstream file;

    file.open(filename + "_EnstrophyData.dat" );

    for( auto& EKin : this->enstrophyTimeSeries )
        file << EKin << std::endl;

    file.close();

    *logging::out << logging::Logger::INFO_INTERMEDIATE << "done!\n";
}


