#include "hip/hip_runtime.h"
//  _    ___      __              __________      _     __        ______________   __
// | |  / (_)____/ /___  ______ _/ / ____/ /_  __(_)___/ /____   /  ___/ __  / /  / /
// | | / / / ___/ __/ / / / __ `/ / /_  / / / / / / __  / ___/  / /___/ /_/ / /  / /
// | |/ / / /  / /_/ /_/ / /_/ / / __/ / / /_/ / / /_/ (__  )  / /_) / ____/ /__/ / 
// |___/_/_/   \__/\__,_/\__,_/_/_/   /_/\__,_/_/\__,_/____/   \____/_/    \_____/
//
//////////////////////////////////////////////////////////////////////////

/* Device code */
#include "LBM/D3Q27.h"
#include "GPU/constant.h"

/////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QVelDeviceCompThinWallsPartOne27(
	real* vx,
	real* vy,
	real* vz,
	real* DD, 
	int* k_Q, 
	real* QQ,
	uint sizeQ,
	int kQ, 
	real om1, 
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	uint size_Mat, 
	bool evenOrOdd)
{
   Distributions27 D;
   if (evenOrOdd==true)
   {
      D.f[dirE   ] = &DD[dirE   *size_Mat];
      D.f[dirW   ] = &DD[dirW   *size_Mat];
      D.f[dirN   ] = &DD[dirN   *size_Mat];
      D.f[dirS   ] = &DD[dirS   *size_Mat];
      D.f[dirT   ] = &DD[dirT   *size_Mat];
      D.f[dirB   ] = &DD[dirB   *size_Mat];
      D.f[dirNE  ] = &DD[dirNE  *size_Mat];
      D.f[dirSW  ] = &DD[dirSW  *size_Mat];
      D.f[dirSE  ] = &DD[dirSE  *size_Mat];
      D.f[dirNW  ] = &DD[dirNW  *size_Mat];
      D.f[dirTE  ] = &DD[dirTE  *size_Mat];
      D.f[dirBW  ] = &DD[dirBW  *size_Mat];
      D.f[dirBE  ] = &DD[dirBE  *size_Mat];
      D.f[dirTW  ] = &DD[dirTW  *size_Mat];
      D.f[dirTN  ] = &DD[dirTN  *size_Mat];
      D.f[dirBS  ] = &DD[dirBS  *size_Mat];
      D.f[dirBN  ] = &DD[dirBN  *size_Mat];
      D.f[dirTS  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirTNE *size_Mat];
      D.f[dirTSW ] = &DD[dirTSW *size_Mat];
      D.f[dirTSE ] = &DD[dirTSE *size_Mat];
      D.f[dirTNW ] = &DD[dirTNW *size_Mat];
      D.f[dirBNE ] = &DD[dirBNE *size_Mat];
      D.f[dirBSW ] = &DD[dirBSW *size_Mat];
      D.f[dirBSE ] = &DD[dirBSE *size_Mat];
      D.f[dirBNW ] = &DD[dirBNW *size_Mat];
   } 
   else
   {
      D.f[dirW   ] = &DD[dirE   *size_Mat];
      D.f[dirE   ] = &DD[dirW   *size_Mat];
      D.f[dirS   ] = &DD[dirN   *size_Mat];
      D.f[dirN   ] = &DD[dirS   *size_Mat];
      D.f[dirB   ] = &DD[dirT   *size_Mat];
      D.f[dirT   ] = &DD[dirB   *size_Mat];
      D.f[dirSW  ] = &DD[dirNE  *size_Mat];
      D.f[dirNE  ] = &DD[dirSW  *size_Mat];
      D.f[dirNW  ] = &DD[dirSE  *size_Mat];
      D.f[dirSE  ] = &DD[dirNW  *size_Mat];
      D.f[dirBW  ] = &DD[dirTE  *size_Mat];
      D.f[dirTE  ] = &DD[dirBW  *size_Mat];
      D.f[dirTW  ] = &DD[dirBE  *size_Mat];
      D.f[dirBE  ] = &DD[dirTW  *size_Mat];
      D.f[dirBS  ] = &DD[dirTN  *size_Mat];
      D.f[dirTN  ] = &DD[dirBS  *size_Mat];
      D.f[dirTS  ] = &DD[dirBN  *size_Mat];
      D.f[dirBN  ] = &DD[dirTS  *size_Mat];
      D.f[dirZERO] = &DD[dirZERO*size_Mat];
      D.f[dirTNE ] = &DD[dirBSW *size_Mat];
      D.f[dirTSW ] = &DD[dirBNE *size_Mat];
      D.f[dirTSE ] = &DD[dirBNW *size_Mat];
      D.f[dirTNW ] = &DD[dirBSE *size_Mat];
      D.f[dirBNE ] = &DD[dirTSW *size_Mat];
      D.f[dirBSW ] = &DD[dirTNE *size_Mat];
      D.f[dirBSE ] = &DD[dirTNW *size_Mat];
      D.f[dirBNW ] = &DD[dirTSE *size_Mat];
   }
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<kQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real VeloX = vx[k];
      real VeloY = vy[k];
      real VeloZ = vz[k]; 
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      uint KQK  = k_Q[k];
      uint kzero= KQK;
      uint ke   = KQK;
      uint kw   = neighborX[KQK];
      uint kn   = KQK;
      uint ks   = neighborY[KQK];
      uint kt   = KQK;
      uint kb   = neighborZ[KQK];
      uint ksw  = neighborY[kw];
      uint kne  = KQK;
      uint kse  = ks;
      uint knw  = kw;
      uint kbw  = neighborZ[kw];
      uint kte  = KQK;
      uint kbe  = kb;
      uint ktw  = kw;
      uint kbs  = neighborZ[ks];
      uint ktn  = KQK;
      uint kbn  = kb;
      uint kts  = ks;
      uint ktse = ks;
      uint kbnw = kbw;
      uint ktnw = kw;
      uint kbse = kbs;
      uint ktsw = ksw;
      uint kbne = kb;
      uint ktne = KQK;
      uint kbsw = neighborZ[ksw];
      ////////////////////////////////////////////////////////////////////////////////
      real f_E,  f_W,  f_N,  f_S,  f_T,  f_B,   f_NE,  f_SW,  f_SE,  f_NW,  f_TE,  f_BW,  f_BE,
         f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

      f_W    = (D.f[dirE   ])[ke   ];
      f_E    = (D.f[dirW   ])[kw   ];
      f_S    = (D.f[dirN   ])[kn   ];
      f_N    = (D.f[dirS   ])[ks   ];
      f_B    = (D.f[dirT   ])[kt   ];
      f_T    = (D.f[dirB   ])[kb   ];
      f_SW   = (D.f[dirNE  ])[kne  ];
      f_NE   = (D.f[dirSW  ])[ksw  ];
      f_NW   = (D.f[dirSE  ])[kse  ];
      f_SE   = (D.f[dirNW  ])[knw  ];
      f_BW   = (D.f[dirTE  ])[kte  ];
      f_TE   = (D.f[dirBW  ])[kbw  ];
      f_TW   = (D.f[dirBE  ])[kbe  ];
      f_BE   = (D.f[dirTW  ])[ktw  ];
      f_BS   = (D.f[dirTN  ])[ktn  ];
      f_TN   = (D.f[dirBS  ])[kbs  ];
      f_TS   = (D.f[dirBN  ])[kbn  ];
      f_BN   = (D.f[dirTS  ])[kts  ];
      f_BSW  = (D.f[dirTNE ])[ktne ];
      f_BNE  = (D.f[dirTSW ])[ktsw ];
      f_BNW  = (D.f[dirTSE ])[ktse ];
      f_BSE  = (D.f[dirTNW ])[ktnw ];
      f_TSW  = (D.f[dirBNE ])[kbne ];
      f_TNE  = (D.f[dirBSW ])[kbsw ];
      f_TNW  = (D.f[dirBSE ])[kbse ];
      f_TSE  = (D.f[dirBNW ])[kbnw ];
      ////////////////////////////////////////////////////////////////////////////////
      real vx1, vx2, vx3, drho, feq, q;
      drho   =  f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
                f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW + 
                f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]); 

      vx1    =  (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                ((f_BE - f_TW)   + (f_TE - f_BW))   + ((f_SE - f_NW)   + (f_NE - f_SW)) +
                (f_E - f_W)) / (one + drho); 
         

      vx2    =   ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
                 ((f_BN - f_TS)   + (f_TN - f_BS))    + (-(f_SE - f_NW)  + (f_NE - f_SW)) +
                 (f_N - f_S)) / (one + drho); 

      vx3    =   (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
                 (-(f_BN - f_TS)  + (f_TN - f_BS))   + ((f_TE - f_BW)   - (f_BE - f_TW)) +
                 (f_T - f_B)) / (one + drho); 

      real cu_sq=c3o2*(vx1*vx1+vx2*vx2+vx3*vx3) * (one + drho);

      //////////////////////////////////////////////////////////////////////////

      q = q_dirE[k];
      if (q>=zero && q<=one)
      {
         feq=c2over27* (drho + c9over2 * ( vx1        )*( vx1        ) * (one + drho)-cu_sq);
		 (D.f[dirW])[kw] = (one - q) / (one + q)*(f_E - f_W + (f_E + f_W - two*feq*om1) / (one - om1))*c1o2 + (q*(f_E + f_W) - six*c2over27*(VeloX)) / (one + q);
	  }

	  q = q_dirW[k];
	  if (q >= zero && q <= one)
	  {
		  feq = c2over27* (drho + c9over2 * (-vx1)*(-vx1) * (one + drho) - cu_sq);
		  (D.f[dirE])[ke] = (one - q) / (one + q)*(f_W - f_E + (f_W + f_E - two*feq*om1) / (one - om1))*c1o2 + (q*(f_W + f_E) - six*c2over27*(-VeloX)) / (one + q);
	  }

	  q = q_dirN[k];
	  if (q >= zero && q <= one)
	  {
		  feq = c2over27* (drho + c9over2 * (vx2)*(vx2) * (one + drho) - cu_sq);
		  (D.f[dirS])[ks] = (one - q) / (one + q)*(f_N - f_S + (f_N + f_S - two*feq*om1) / (one - om1))*c1o2 + (q*(f_N + f_S) - six*c2over27*(VeloY)) / (one + q);
	  }

	  q = q_dirS[k];
	  if (q >= zero && q <= one)
	  {
		  feq = c2over27* (drho + c9over2 * (-vx2)*(-vx2) * (one + drho) - cu_sq);
		  (D.f[dirN])[kn] = (one - q) / (one + q)*(f_S - f_N + (f_S + f_N - two*feq*om1) / (one - om1))*c1o2 + (q*(f_S + f_N) - six*c2over27*(-VeloY)) / (one + q);
	  }

	  q = q_dirT[k];
	  if (q >= zero && q <= one)
	  {
		  feq = c2over27* (drho + c9over2 * (vx3)*(vx3) * (one + drho) - cu_sq);
		  (D.f[dirB])[kb] = (one - q) / (one + q)*(f_T - f_B + (f_T + f_B - two*feq*om1) / (one - om1))*c1o2 + (q*(f_T + f_B) - six*c2over27*(VeloZ)) / (one + q);
	  }

	  q = q_dirB[k];
	  if (q >= zero && q <= one)
	  {
		  feq = c2over27* (drho + c9over2 * (-vx3)*(-vx3) * (one + drho) - cu_sq);
		  (D.f[dirT])[kt] = (one - q) / (one + q)*(f_B - f_T + (f_B + f_T - two*feq*om1) / (one - om1))*c1o2 + (q*(f_B + f_T) - six*c2over27*(-VeloZ)) / (one + q);
      }

      q = q_dirNE[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * ( vx1+vx2    )*( vx1+vx2    ) * (one + drho)-cu_sq);
         (D.f[dirSW])[ksw]=(one-q)/(one+q)*(f_NE-f_SW+(f_NE+f_SW-two*feq*om1)/(one-om1))*c1o2+(q*(f_NE+f_SW)-six*c1over54*( VeloX+VeloY))/(one+q);
      }

      q = q_dirSW[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * (-vx1-vx2    )*(-vx1-vx2    ) * (one + drho)-cu_sq);
         (D.f[dirNE])[kne]=(one-q)/(one+q)*(f_SW-f_NE+(f_SW+f_NE-two*feq*om1)/(one-om1))*c1o2+(q*(f_SW+f_NE)-six*c1over54*(-VeloX-VeloY))/(one+q);
      }

      q = q_dirSE[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * ( vx1-vx2    )*( vx1-vx2    ) * (one + drho)-cu_sq);
         (D.f[dirNW])[knw]=(one-q)/(one+q)*(f_SE-f_NW+(f_SE+f_NW-two*feq*om1)/(one-om1))*c1o2+(q*(f_SE+f_NW)-six*c1over54*( VeloX-VeloY))/(one+q);
      }

      q = q_dirNW[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * (-vx1+vx2    )*(-vx1+vx2    ) * (one + drho)-cu_sq);
         (D.f[dirSE])[kse]=(one-q)/(one+q)*(f_NW-f_SE+(f_NW+f_SE-two*feq*om1)/(one-om1))*c1o2+(q*(f_NW+f_SE)-six*c1over54*(-VeloX+VeloY))/(one+q);
      }

      q = q_dirTE[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * ( vx1    +vx3)*( vx1    +vx3) * (one + drho)-cu_sq);
         (D.f[dirBW])[kbw]=(one-q)/(one+q)*(f_TE-f_BW+(f_TE+f_BW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TE+f_BW)-six*c1over54*( VeloX+VeloZ))/(one+q);
      }

      q = q_dirBW[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * (-vx1    -vx3)*(-vx1    -vx3) * (one + drho)-cu_sq);
         (D.f[dirTE])[kte]=(one-q)/(one+q)*(f_BW-f_TE+(f_BW+f_TE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BW+f_TE)-six*c1over54*(-VeloX-VeloZ))/(one+q);
      }

      q = q_dirBE[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * ( vx1    -vx3)*( vx1    -vx3) * (one + drho)-cu_sq);
         (D.f[dirTW])[ktw]=(one-q)/(one+q)*(f_BE-f_TW+(f_BE+f_TW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BE+f_TW)-six*c1over54*( VeloX-VeloZ))/(one+q);
      }

      q = q_dirTW[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * (-vx1    +vx3)*(-vx1    +vx3) * (one + drho)-cu_sq);
         (D.f[dirBE])[kbe]=(one-q)/(one+q)*(f_TW-f_BE+(f_TW+f_BE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TW+f_BE)-six*c1over54*(-VeloX+VeloZ))/(one+q);
      }

      q = q_dirTN[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * (     vx2+vx3)*(     vx2+vx3) * (one + drho)-cu_sq);
         (D.f[dirBS])[kbs]=(one-q)/(one+q)*(f_TN-f_BS+(f_TN+f_BS-two*feq*om1)/(one-om1))*c1o2+(q*(f_TN+f_BS)-six*c1over54*( VeloY+VeloZ))/(one+q);
      }

      q = q_dirBS[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * (    -vx2-vx3)*(    -vx2-vx3) * (one + drho)-cu_sq);
         (D.f[dirTN])[ktn]=(one-q)/(one+q)*(f_BS-f_TN+(f_BS+f_TN-two*feq*om1)/(one-om1))*c1o2+(q*(f_BS+f_TN)-six*c1over54*(-VeloY-VeloZ))/(one+q);
      }

      q = q_dirBN[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * (     vx2-vx3)*(     vx2-vx3) * (one + drho)-cu_sq);
         (D.f[dirTS])[kts]=(one-q)/(one+q)*(f_BN-f_TS+(f_BN+f_TS-two*feq*om1)/(one-om1))*c1o2+(q*(f_BN+f_TS)-six*c1over54*( VeloY-VeloZ))/(one+q);
      }

      q = q_dirTS[k];
      if (q>=zero && q<=one)
      {
         feq=c1over54* (drho + c9over2 * (    -vx2+vx3)*(    -vx2+vx3) * (one + drho)-cu_sq);
         (D.f[dirBN])[kbn]=(one-q)/(one+q)*(f_TS-f_BN+(f_TS+f_BN-two*feq*om1)/(one-om1))*c1o2+(q*(f_TS+f_BN)-six*c1over54*(-VeloY+VeloZ))/(one+q);
      }

      q = q_dirTNE[k];
      if (q>=zero && q<=one)
      {
         feq=c1over216*(drho + c9over2 * ( vx1+vx2+vx3)*( vx1+vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBSW])[kbsw]=(one-q)/(one+q)*(f_TNE-f_BSW+(f_TNE+f_BSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNE+f_BSW)-six*c1over216*( VeloX+VeloY+VeloZ))/(one+q);
      }

      q = q_dirBSW[k];
      if (q>=zero && q<=one)
      {
         feq=c1over216*(drho + c9over2 * (-vx1-vx2-vx3)*(-vx1-vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTNE])[ktne]=(one-q)/(one+q)*(f_BSW-f_TNE+(f_BSW+f_TNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSW+f_TNE)-six*c1over216*(-VeloX-VeloY-VeloZ))/(one+q);
      }

      q = q_dirBNE[k];
      if (q>=zero && q<=one)
      {
         feq=c1over216*(drho + c9over2 * ( vx1+vx2-vx3)*( vx1+vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTSW])[ktsw]=(one-q)/(one+q)*(f_BNE-f_TSW+(f_BNE+f_TSW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNE+f_TSW)-six*c1over216*( VeloX+VeloY-VeloZ))/(one+q);
      }

      q = q_dirTSW[k];
      if (q>=zero && q<=one)
      {
         feq=c1over216*(drho + c9over2 * (-vx1-vx2+vx3)*(-vx1-vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBNE])[kbne]=(one-q)/(one+q)*(f_TSW-f_BNE+(f_TSW+f_BNE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSW+f_BNE)-six*c1over216*(-VeloX-VeloY+VeloZ))/(one+q);
      }

      q = q_dirTSE[k];
      if (q>=zero && q<=one)
      {
         feq=c1over216*(drho + c9over2 * ( vx1-vx2+vx3)*( vx1-vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBNW])[kbnw]=(one-q)/(one+q)*(f_TSE-f_BNW+(f_TSE+f_BNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_TSE+f_BNW)-six*c1over216*( VeloX-VeloY+VeloZ))/(one+q);
      }

      q = q_dirBNW[k];
      if (q>=zero && q<=one)
      {
         feq=c1over216*(drho + c9over2 * (-vx1+vx2-vx3)*(-vx1+vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTSE])[ktse]=(one-q)/(one+q)*(f_BNW-f_TSE+(f_BNW+f_TSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_BNW+f_TSE)-six*c1over216*(-VeloX+VeloY-VeloZ))/(one+q);
      }

      q = q_dirBSE[k];
      if (q>=zero && q<=one)
      {
         feq=c1over216*(drho + c9over2 * ( vx1-vx2-vx3)*( vx1-vx2-vx3) * (one + drho)-cu_sq); 
         (D.f[dirTNW])[ktnw]=(one-q)/(one+q)*(f_BSE-f_TNW+(f_BSE+f_TNW-two*feq*om1)/(one-om1))*c1o2+(q*(f_BSE+f_TNW)-six*c1over216*( VeloX-VeloY-VeloZ))/(one+q);
      }

      q = q_dirTNW[k];
      if (q>=zero && q<=one)
      {
         feq=c1over216*(drho + c9over2 * (-vx1+vx2+vx3)*(-vx1+vx2+vx3) * (one + drho)-cu_sq); 
         (D.f[dirBSE])[kbse]=(one-q)/(one+q)*(f_TNW-f_BSE+(f_TNW+f_BSE-two*feq*om1)/(one-om1))*c1o2+(q*(f_TNW+f_BSE)-six*c1over216*(-VeloX+VeloY+VeloZ))/(one+q);
      }
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QDeviceCompThinWallsPartOne27(
	real* DD,
	int* k_Q,
	real* QQ,
	unsigned int sizeQ,
	int kQ,
	real om1,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	unsigned int size_Mat,
	bool evenOrOdd)
{
	Distributions27 D;
	if (evenOrOdd == true)
	{
		D.f[dirE] = &DD[dirE   *size_Mat];
		D.f[dirW] = &DD[dirW   *size_Mat];
		D.f[dirN] = &DD[dirN   *size_Mat];
		D.f[dirS] = &DD[dirS   *size_Mat];
		D.f[dirT] = &DD[dirT   *size_Mat];
		D.f[dirB] = &DD[dirB   *size_Mat];
		D.f[dirNE] = &DD[dirNE  *size_Mat];
		D.f[dirSW] = &DD[dirSW  *size_Mat];
		D.f[dirSE] = &DD[dirSE  *size_Mat];
		D.f[dirNW] = &DD[dirNW  *size_Mat];
		D.f[dirTE] = &DD[dirTE  *size_Mat];
		D.f[dirBW] = &DD[dirBW  *size_Mat];
		D.f[dirBE] = &DD[dirBE  *size_Mat];
		D.f[dirTW] = &DD[dirTW  *size_Mat];
		D.f[dirTN] = &DD[dirTN  *size_Mat];
		D.f[dirBS] = &DD[dirBS  *size_Mat];
		D.f[dirBN] = &DD[dirBN  *size_Mat];
		D.f[dirTS] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE] = &DD[dirTNE *size_Mat];
		D.f[dirTSW] = &DD[dirTSW *size_Mat];
		D.f[dirTSE] = &DD[dirTSE *size_Mat];
		D.f[dirTNW] = &DD[dirTNW *size_Mat];
		D.f[dirBNE] = &DD[dirBNE *size_Mat];
		D.f[dirBSW] = &DD[dirBSW *size_Mat];
		D.f[dirBSE] = &DD[dirBSE *size_Mat];
		D.f[dirBNW] = &DD[dirBNW *size_Mat];
	}
	else
	{
		D.f[dirW] = &DD[dirE   *size_Mat];
		D.f[dirE] = &DD[dirW   *size_Mat];
		D.f[dirS] = &DD[dirN   *size_Mat];
		D.f[dirN] = &DD[dirS   *size_Mat];
		D.f[dirB] = &DD[dirT   *size_Mat];
		D.f[dirT] = &DD[dirB   *size_Mat];
		D.f[dirSW] = &DD[dirNE  *size_Mat];
		D.f[dirNE] = &DD[dirSW  *size_Mat];
		D.f[dirNW] = &DD[dirSE  *size_Mat];
		D.f[dirSE] = &DD[dirNW  *size_Mat];
		D.f[dirBW] = &DD[dirTE  *size_Mat];
		D.f[dirTE] = &DD[dirBW  *size_Mat];
		D.f[dirTW] = &DD[dirBE  *size_Mat];
		D.f[dirBE] = &DD[dirTW  *size_Mat];
		D.f[dirBS] = &DD[dirTN  *size_Mat];
		D.f[dirTN] = &DD[dirBS  *size_Mat];
		D.f[dirTS] = &DD[dirBN  *size_Mat];
		D.f[dirBN] = &DD[dirTS  *size_Mat];
		D.f[dirZERO] = &DD[dirZERO*size_Mat];
		D.f[dirTNE] = &DD[dirBSW *size_Mat];
		D.f[dirTSW] = &DD[dirBNE *size_Mat];
		D.f[dirTSE] = &DD[dirBNW *size_Mat];
		D.f[dirTNW] = &DD[dirBSE *size_Mat];
		D.f[dirBNE] = &DD[dirTSW *size_Mat];
		D.f[dirBSW] = &DD[dirTNE *size_Mat];
		D.f[dirBSE] = &DD[dirTNW *size_Mat];
		D.f[dirBNW] = &DD[dirTSE *size_Mat];
	}
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k < kQ)
	{
		////////////////////////////////////////////////////////////////////////////////
		real *q_dirE, *q_dirW, *q_dirN, *q_dirS, *q_dirT, *q_dirB,
			*q_dirNE, *q_dirSW, *q_dirSE, *q_dirNW, *q_dirTE, *q_dirBW,
			*q_dirBE, *q_dirTW, *q_dirTN, *q_dirBS, *q_dirBN, *q_dirTS,
			*q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
			*q_dirBSE, *q_dirBNW;
		q_dirE = &QQ[dirE   *sizeQ];
		q_dirW = &QQ[dirW   *sizeQ];
		q_dirN = &QQ[dirN   *sizeQ];
		q_dirS = &QQ[dirS   *sizeQ];
		q_dirT = &QQ[dirT   *sizeQ];
		q_dirB = &QQ[dirB   *sizeQ];
		q_dirNE = &QQ[dirNE  *sizeQ];
		q_dirSW = &QQ[dirSW  *sizeQ];
		q_dirSE = &QQ[dirSE  *sizeQ];
		q_dirNW = &QQ[dirNW  *sizeQ];
		q_dirTE = &QQ[dirTE  *sizeQ];
		q_dirBW = &QQ[dirBW  *sizeQ];
		q_dirBE = &QQ[dirBE  *sizeQ];
		q_dirTW = &QQ[dirTW  *sizeQ];
		q_dirTN = &QQ[dirTN  *sizeQ];
		q_dirBS = &QQ[dirBS  *sizeQ];
		q_dirBN = &QQ[dirBN  *sizeQ];
		q_dirTS = &QQ[dirTS  *sizeQ];
		q_dirTNE = &QQ[dirTNE *sizeQ];
		q_dirTSW = &QQ[dirTSW *sizeQ];
		q_dirTSE = &QQ[dirTSE *sizeQ];
		q_dirTNW = &QQ[dirTNW *sizeQ];
		q_dirBNE = &QQ[dirBNE *sizeQ];
		q_dirBSW = &QQ[dirBSW *sizeQ];
		q_dirBSE = &QQ[dirBSE *sizeQ];
		q_dirBNW = &QQ[dirBNW *sizeQ];
		////////////////////////////////////////////////////////////////////////////////
		//index
		unsigned int KQK = k_Q[k];
		unsigned int kzero = KQK;
		unsigned int ke = KQK;
		unsigned int kw = neighborX[KQK];
		unsigned int kn = KQK;
		unsigned int ks = neighborY[KQK];
		unsigned int kt = KQK;
		unsigned int kb = neighborZ[KQK];
		unsigned int ksw = neighborY[kw];
		unsigned int kne = KQK;
		unsigned int kse = ks;
		unsigned int knw = kw;
		unsigned int kbw = neighborZ[kw];
		unsigned int kte = KQK;
		unsigned int kbe = kb;
		unsigned int ktw = kw;
		unsigned int kbs = neighborZ[ks];
		unsigned int ktn = KQK;
		unsigned int kbn = kb;
		unsigned int kts = ks;
		unsigned int ktse = ks;
		unsigned int kbnw = kbw;
		unsigned int ktnw = kw;
		unsigned int kbse = kbs;
		unsigned int ktsw = ksw;
		unsigned int kbne = kb;
		unsigned int ktne = KQK;
		unsigned int kbsw = neighborZ[ksw];
		////////////////////////////////////////////////////////////////////////////////
		real f_E, f_W, f_N, f_S, f_T, f_B, f_NE, f_SW, f_SE, f_NW, f_TE, f_BW, f_BE,
			f_TW, f_TN, f_BS, f_BN, f_TS, f_TNE, f_TSW, f_TSE, f_TNW, f_BNE, f_BSW, f_BSE, f_BNW;

		f_W = (D.f[dirE])[ke];
		f_E = (D.f[dirW])[kw];
		f_S = (D.f[dirN])[kn];
		f_N = (D.f[dirS])[ks];
		f_B = (D.f[dirT])[kt];
		f_T = (D.f[dirB])[kb];
		f_SW = (D.f[dirNE])[kne];
		f_NE = (D.f[dirSW])[ksw];
		f_NW = (D.f[dirSE])[kse];
		f_SE = (D.f[dirNW])[knw];
		f_BW = (D.f[dirTE])[kte];
		f_TE = (D.f[dirBW])[kbw];
		f_TW = (D.f[dirBE])[kbe];
		f_BE = (D.f[dirTW])[ktw];
		f_BS = (D.f[dirTN])[ktn];
		f_TN = (D.f[dirBS])[kbs];
		f_TS = (D.f[dirBN])[kbn];
		f_BN = (D.f[dirTS])[kts];
		f_BSW = (D.f[dirTNE])[ktne];
		f_BNE = (D.f[dirTSW])[ktsw];
		f_BNW = (D.f[dirTSE])[ktse];
		f_BSE = (D.f[dirTNW])[ktnw];
		f_TSW = (D.f[dirBNE])[kbne];
		f_TNE = (D.f[dirBSW])[kbsw];
		f_TNW = (D.f[dirBSE])[kbse];
		f_TSE = (D.f[dirBNW])[kbnw];
		////////////////////////////////////////////////////////////////////////////////
		real vx1, vx2, vx3, drho, feq, q;
		drho = f_TSE + f_TNW + f_TNE + f_TSW + f_BSE + f_BNW + f_BNE + f_BSW +
			f_BN + f_TS + f_TN + f_BS + f_BE + f_TW + f_TE + f_BW + f_SE + f_NW + f_NE + f_SW +
			f_T + f_B + f_N + f_S + f_E + f_W + ((D.f[dirZERO])[kzero]);

		vx1 = (((f_TSE - f_BNW) - (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
			((f_BE - f_TW) + (f_TE - f_BW)) + ((f_SE - f_NW) + (f_NE - f_SW)) +
			(f_E - f_W)) / (one + drho);


		vx2 = ((-(f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) - (f_TSW - f_BNE)) +
			((f_BN - f_TS) + (f_TN - f_BS)) + (-(f_SE - f_NW) + (f_NE - f_SW)) +
			(f_N - f_S)) / (one + drho);

		vx3 = (((f_TSE - f_BNW) + (f_TNW - f_BSE)) + ((f_TNE - f_BSW) + (f_TSW - f_BNE)) +
			(-(f_BN - f_TS) + (f_TN - f_BS)) + ((f_TE - f_BW) - (f_BE - f_TW)) +
			(f_T - f_B)) / (one + drho);

		////////////////////////////////////////////////////////////////////////////////
		real cu_sq = c3o2*(vx1*vx1 + vx2*vx2 + vx3*vx3) * (one + drho);
		////////////////////////////////////////////////////////////////////////////////

		q = q_dirE[k];
		if (q >= zero && q <= one)
		{
			feq = c2over27* (drho + c9over2*(vx1)*(vx1) * (one + drho) - cu_sq);
			(D.f[dirW])[kw] = (one - q) / (one + q)*(f_E - f_W + (f_E + f_W - two*feq*om1) / (one - om1))*c1o2 + (q*(f_E + f_W)) / (one + q);
		}

		q = q_dirW[k];
		if (q >= zero && q <= one)
		{
			feq = c2over27* (drho + c9over2*(-vx1)*(-vx1) * (one + drho) - cu_sq);
			(D.f[dirE])[ke] = (one - q) / (one + q)*(f_W - f_E + (f_W + f_E - two*feq*om1) / (one - om1))*c1o2 + (q*(f_W + f_E)) / (one + q);
		}

		q = q_dirN[k];
		if (q >= zero && q <= one)
		{
			feq = c2over27* (drho + c9over2*(vx2)*(vx2) * (one + drho) - cu_sq);
			(D.f[dirS])[ks] = (one - q) / (one + q)*(f_N - f_S + (f_N + f_S - two*feq*om1) / (one - om1))*c1o2 + (q*(f_N + f_S)) / (one + q);
		}

		q = q_dirS[k];
		if (q >= zero && q <= one)
		{
			feq = c2over27* (drho + c9over2*(-vx2)*(-vx2) * (one + drho) - cu_sq);
			(D.f[dirN])[kn] = (one - q) / (one + q)*(f_S - f_N + (f_S + f_N - two*feq*om1) / (one - om1))*c1o2 + (q*(f_S + f_N)) / (one + q);
		}

		q = q_dirT[k];
		if (q >= zero && q <= one)
		{
			feq = c2over27* (drho + c9over2*(vx3)*(vx3) * (one + drho) - cu_sq);
			(D.f[dirB])[kb] = (one - q) / (one + q)*(f_T - f_B + (f_T + f_B - two*feq*om1) / (one - om1))*c1o2 + (q*(f_T + f_B)) / (one + q);
		}

		q = q_dirB[k];
		if (q >= zero && q <= one)
		{
			feq = c2over27* (drho + c9over2*(-vx3)*(-vx3) * (one + drho) - cu_sq);
			(D.f[dirT])[kt] = (one - q) / (one + q)*(f_B - f_T + (f_B + f_T - two*feq*om1) / (one - om1))*c1o2 + (q*(f_B + f_T)) / (one + q);
		}

		q = q_dirNE[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(vx1 + vx2)*(vx1 + vx2) * (one + drho) - cu_sq);
			(D.f[dirSW])[ksw] = (one - q) / (one + q)*(f_NE - f_SW + (f_NE + f_SW - two*feq*om1) / (one - om1))*c1o2 + (q*(f_NE + f_SW)) / (one + q);
		}

		q = q_dirSW[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(-vx1 - vx2)*(-vx1 - vx2) * (one + drho) - cu_sq);
			(D.f[dirNE])[kne] = (one - q) / (one + q)*(f_SW - f_NE + (f_SW + f_NE - two*feq*om1) / (one - om1))*c1o2 + (q*(f_SW + f_NE)) / (one + q);
		}

		q = q_dirSE[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(vx1 - vx2)*(vx1 - vx2) * (one + drho) - cu_sq);
			(D.f[dirNW])[knw] = (one - q) / (one + q)*(f_SE - f_NW + (f_SE + f_NW - two*feq*om1) / (one - om1))*c1o2 + (q*(f_SE + f_NW)) / (one + q);
		}

		q = q_dirNW[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(-vx1 + vx2)*(-vx1 + vx2) * (one + drho) - cu_sq);
			(D.f[dirSE])[kse] = (one - q) / (one + q)*(f_NW - f_SE + (f_NW + f_SE - two*feq*om1) / (one - om1))*c1o2 + (q*(f_NW + f_SE)) / (one + q);
		}

		q = q_dirTE[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(vx1 + vx3)*(vx1 + vx3) * (one + drho) - cu_sq);
			(D.f[dirBW])[kbw] = (one - q) / (one + q)*(f_TE - f_BW + (f_TE + f_BW - two*feq*om1) / (one - om1))*c1o2 + (q*(f_TE + f_BW)) / (one + q);
		}

		q = q_dirBW[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(-vx1 - vx3)*(-vx1 - vx3) * (one + drho) - cu_sq);
			(D.f[dirTE])[kte] = (one - q) / (one + q)*(f_BW - f_TE + (f_BW + f_TE - two*feq*om1) / (one - om1))*c1o2 + (q*(f_BW + f_TE)) / (one + q);
		}

		q = q_dirBE[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(vx1 - vx3)*(vx1 - vx3) * (one + drho) - cu_sq);
			(D.f[dirTW])[ktw] = (one - q) / (one + q)*(f_BE - f_TW + (f_BE + f_TW - two*feq*om1) / (one - om1))*c1o2 + (q*(f_BE + f_TW)) / (one + q);
		}

		q = q_dirTW[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(-vx1 + vx3)*(-vx1 + vx3) * (one + drho) - cu_sq);
			(D.f[dirBE])[kbe] = (one - q) / (one + q)*(f_TW - f_BE + (f_TW + f_BE - two*feq*om1) / (one - om1))*c1o2 + (q*(f_TW + f_BE)) / (one + q);
		}

		q = q_dirTN[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(vx2 + vx3)*(vx2 + vx3) * (one + drho) - cu_sq);
			(D.f[dirBS])[kbs] = (one - q) / (one + q)*(f_TN - f_BS + (f_TN + f_BS - two*feq*om1) / (one - om1))*c1o2 + (q*(f_TN + f_BS)) / (one + q);
		}

		q = q_dirBS[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(-vx2 - vx3)*(-vx2 - vx3) * (one + drho) - cu_sq);
			(D.f[dirTN])[ktn] = (one - q) / (one + q)*(f_BS - f_TN + (f_BS + f_TN - two*feq*om1) / (one - om1))*c1o2 + (q*(f_BS + f_TN)) / (one + q);
		}

		q = q_dirBN[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(vx2 - vx3)*(vx2 - vx3) * (one + drho) - cu_sq);
			(D.f[dirTS])[kts] = (one - q) / (one + q)*(f_BN - f_TS + (f_BN + f_TS - two*feq*om1) / (one - om1))*c1o2 + (q*(f_BN + f_TS)) / (one + q);
		}

		q = q_dirTS[k];
		if (q >= zero && q <= one)
		{
			feq = c1over54* (drho + c9over2*(-vx2 + vx3)*(-vx2 + vx3) * (one + drho) - cu_sq);
			(D.f[dirBN])[kbn] = (one - q) / (one + q)*(f_TS - f_BN + (f_TS + f_BN - two*feq*om1) / (one - om1))*c1o2 + (q*(f_TS + f_BN)) / (one + q);
		}

		q = q_dirTNE[k];
		if (q >= zero && q <= one)
		{
			feq = c1over216*(drho + c9over2*(vx1 + vx2 + vx3)*(vx1 + vx2 + vx3) * (one + drho) - cu_sq);
			(D.f[dirBSW])[kbsw] = (one - q) / (one + q)*(f_TNE - f_BSW + (f_TNE + f_BSW - two*feq*om1) / (one - om1))*c1o2 + (q*(f_TNE + f_BSW)) / (one + q);
		}

		q = q_dirBSW[k];
		if (q >= zero && q <= one)
		{
			feq = c1over216*(drho + c9over2*(-vx1 - vx2 - vx3)*(-vx1 - vx2 - vx3) * (one + drho) - cu_sq);
			(D.f[dirTNE])[ktne] = (one - q) / (one + q)*(f_BSW - f_TNE + (f_BSW + f_TNE - two*feq*om1) / (one - om1))*c1o2 + (q*(f_BSW + f_TNE)) / (one + q);
		}

		q = q_dirBNE[k];
		if (q >= zero && q <= one)
		{
			feq = c1over216*(drho + c9over2*(vx1 + vx2 - vx3)*(vx1 + vx2 - vx3) * (one + drho) - cu_sq);
			(D.f[dirTSW])[ktsw] = (one - q) / (one + q)*(f_BNE - f_TSW + (f_BNE + f_TSW - two*feq*om1) / (one - om1))*c1o2 + (q*(f_BNE + f_TSW)) / (one + q);
		}

		q = q_dirTSW[k];
		if (q >= zero && q <= one)
		{
			feq = c1over216*(drho + c9over2*(-vx1 - vx2 + vx3)*(-vx1 - vx2 + vx3) * (one + drho) - cu_sq);
			(D.f[dirBNE])[kbne] = (one - q) / (one + q)*(f_TSW - f_BNE + (f_TSW + f_BNE - two*feq*om1) / (one - om1))*c1o2 + (q*(f_TSW + f_BNE)) / (one + q);
		}

		q = q_dirTSE[k];
		if (q >= zero && q <= one)
		{
			feq = c1over216*(drho + c9over2*(vx1 - vx2 + vx3)*(vx1 - vx2 + vx3) * (one + drho) - cu_sq);
			(D.f[dirBNW])[kbnw] = (one - q) / (one + q)*(f_TSE - f_BNW + (f_TSE + f_BNW - two*feq*om1) / (one - om1))*c1o2 + (q*(f_TSE + f_BNW)) / (one + q);
		}

		q = q_dirBNW[k];
		if (q >= zero && q <= one)
		{
			feq = c1over216*(drho + c9over2*(-vx1 + vx2 - vx3)*(-vx1 + vx2 - vx3) * (one + drho) - cu_sq);
			(D.f[dirTSE])[ktse] = (one - q) / (one + q)*(f_BNW - f_TSE + (f_BNW + f_TSE - two*feq*om1) / (one - om1))*c1o2 + (q*(f_BNW + f_TSE)) / (one + q);
		}

		q = q_dirBSE[k];
		if (q >= zero && q <= one)
		{
			feq = c1over216*(drho + c9over2*(vx1 - vx2 - vx3)*(vx1 - vx2 - vx3) * (one + drho) - cu_sq);
			(D.f[dirTNW])[ktnw] = (one - q) / (one + q)*(f_BSE - f_TNW + (f_BSE + f_TNW - two*feq*om1) / (one - om1))*c1o2 + (q*(f_BSE + f_TNW)) / (one + q);
		}

		q = q_dirTNW[k];
		if (q >= zero && q <= one)
		{
			feq = c1over216*(drho + c9over2*(-vx1 + vx2 + vx3)*(-vx1 + vx2 + vx3) * (one + drho) - cu_sq);
			(D.f[dirBSE])[kbse] = (one - q) / (one + q)*(f_TNW - f_BSE + (f_TNW + f_BSE - two*feq*om1) / (one - om1))*c1o2 + (q*(f_TNW + f_BSE)) / (one + q);
		}
	}
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////







































//////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void QThinWallsPartTwo27(
	real* DD, 
	int* k_Q, 
	real* QQ,
	uint sizeQ,
	int kQ, 
	uint* geom,
	uint* neighborX,
	uint* neighborY,
	uint* neighborZ,
	uint* neighborWSB,
	uint size_Mat, 
	bool evenOrOdd)
{
   ////////////////////////////////////////////////////////////////////////////////
   const unsigned  x = threadIdx.x;  // Globaler x-Index 
   const unsigned  y = blockIdx.x;   // Globaler y-Index 
   const unsigned  z = blockIdx.y;   // Globaler z-Index 

   const unsigned nx = blockDim.x;
   const unsigned ny = gridDim.x;

   const unsigned k = nx*(ny*z + y) + x;
   //////////////////////////////////////////////////////////////////////////

   if(k<kQ)
   {
      ////////////////////////////////////////////////////////////////////////////////
      real *q_dirE,   *q_dirW,   *q_dirN,   *q_dirS,   *q_dirT,   *q_dirB, 
            *q_dirNE,  *q_dirSW,  *q_dirSE,  *q_dirNW,  *q_dirTE,  *q_dirBW,
            *q_dirBE,  *q_dirTW,  *q_dirTN,  *q_dirBS,  *q_dirBN,  *q_dirTS,
            *q_dirTNE, *q_dirTSW, *q_dirTSE, *q_dirTNW, *q_dirBNE, *q_dirBSW,
            *q_dirBSE, *q_dirBNW; 
      q_dirE   = &QQ[dirE   *sizeQ];
      q_dirW   = &QQ[dirW   *sizeQ];
      q_dirN   = &QQ[dirN   *sizeQ];
      q_dirS   = &QQ[dirS   *sizeQ];
      q_dirT   = &QQ[dirT   *sizeQ];
      q_dirB   = &QQ[dirB   *sizeQ];
      q_dirNE  = &QQ[dirNE  *sizeQ];
      q_dirSW  = &QQ[dirSW  *sizeQ];
      q_dirSE  = &QQ[dirSE  *sizeQ];
      q_dirNW  = &QQ[dirNW  *sizeQ];
      q_dirTE  = &QQ[dirTE  *sizeQ];
      q_dirBW  = &QQ[dirBW  *sizeQ];
      q_dirBE  = &QQ[dirBE  *sizeQ];
      q_dirTW  = &QQ[dirTW  *sizeQ];
      q_dirTN  = &QQ[dirTN  *sizeQ];
      q_dirBS  = &QQ[dirBS  *sizeQ];
      q_dirBN  = &QQ[dirBN  *sizeQ];
      q_dirTS  = &QQ[dirTS  *sizeQ];
      q_dirTNE = &QQ[dirTNE *sizeQ];
      q_dirTSW = &QQ[dirTSW *sizeQ];
      q_dirTSE = &QQ[dirTSE *sizeQ];
      q_dirTNW = &QQ[dirTNW *sizeQ];
      q_dirBNE = &QQ[dirBNE *sizeQ];
      q_dirBSW = &QQ[dirBSW *sizeQ];
      q_dirBSE = &QQ[dirBSE *sizeQ];
      q_dirBNW = &QQ[dirBNW *sizeQ];
      ////////////////////////////////////////////////////////////////////////////////
      //index
      uint KQK  = k_Q[k];
      uint kzero= KQK;
      uint ke   = KQK;
      uint kw   = neighborX[KQK];
      uint kn   = KQK;
      uint ks   = neighborY[KQK];
      uint kt   = KQK;
      uint kb   = neighborZ[KQK];
      uint ksw  = neighborY[kw];
      uint kne  = KQK;
      uint kse  = ks;
      uint knw  = kw;
      uint kbw  = neighborZ[kw];
      uint kte  = KQK;
      uint kbe  = kb;
      uint ktw  = kw;
      uint kbs  = neighborZ[ks];
      uint ktn  = KQK;
      uint kbn  = kb;
      uint kts  = ks;
      uint ktse = ks;
      uint kbnw = kbw;
      uint ktnw = kw;
      uint kbse = kbs;
      uint ktsw = ksw;
      uint kbne = kb;
      uint ktne = KQK;
      uint kbsw = neighborZ[ksw];
	  ////////////////////////////////////////////////////////////////////////////////
	  //anti ET intermediate steps
	  uint kmmm = neighborWSB[KQK]; // -1 -1 -1
	  uint k0mm = neighborX[kmmm];  //  0 -1 -1
	  uint km0m = neighborY[kmmm];  // -1  0 -1
	  uint kmm0 = neighborZ[kmmm];  // -1 -1  0
	  uint k0m0 = neighborX[kmm0];  //  0 -1  0
	  uint km00 = neighborY[kmm0];  // -1  0  0
	  /////////////////////////////////////////////////
	  //final indices for anti ET
	  uint kpmm = neighborX[k0mm];  //  1 -1 -1
	  uint kmpm = neighborY[km0m];  // -1  1 -1
	  uint kmmp = neighborZ[kmm0];  // -1 -1  1
	  uint kmp0 = neighborY[km00];  // -1  1  0
	  uint km0p = neighborZ[km00];  // -1  0  1
	  uint k0mp = neighborZ[k0m0];  //  0 -1  1
	  ////////////////////////////////////////////////////////////////////////////////
	  Distributions27 D, DN;
	  if (evenOrOdd == true)
	  {
		  D.f[dirE] = &DD[dirE   *size_Mat];
		  D.f[dirW] = &DD[dirW   *size_Mat];
		  D.f[dirN] = &DD[dirN   *size_Mat];
		  D.f[dirS] = &DD[dirS   *size_Mat];
		  D.f[dirT] = &DD[dirT   *size_Mat];
		  D.f[dirB] = &DD[dirB   *size_Mat];
		  D.f[dirNE] = &DD[dirNE  *size_Mat];
		  D.f[dirSW] = &DD[dirSW  *size_Mat];
		  D.f[dirSE] = &DD[dirSE  *size_Mat];
		  D.f[dirNW] = &DD[dirNW  *size_Mat];
		  D.f[dirTE] = &DD[dirTE  *size_Mat];
		  D.f[dirBW] = &DD[dirBW  *size_Mat];
		  D.f[dirBE] = &DD[dirBE  *size_Mat];
		  D.f[dirTW] = &DD[dirTW  *size_Mat];
		  D.f[dirTN] = &DD[dirTN  *size_Mat];
		  D.f[dirBS] = &DD[dirBS  *size_Mat];
		  D.f[dirBN] = &DD[dirBN  *size_Mat];
		  D.f[dirTS] = &DD[dirTS  *size_Mat];
		  D.f[dirZERO] = &DD[dirZERO*size_Mat];
		  D.f[dirTNE] = &DD[dirTNE *size_Mat];
		  D.f[dirTSW] = &DD[dirTSW *size_Mat];
		  D.f[dirTSE] = &DD[dirTSE *size_Mat];
		  D.f[dirTNW] = &DD[dirTNW *size_Mat];
		  D.f[dirBNE] = &DD[dirBNE *size_Mat];
		  D.f[dirBSW] = &DD[dirBSW *size_Mat];
		  D.f[dirBSE] = &DD[dirBSE *size_Mat];
		  D.f[dirBNW] = &DD[dirBNW *size_Mat];
	  }
	  else
	  {
		  D.f[dirW] = &DD[dirE   *size_Mat];
		  D.f[dirE] = &DD[dirW   *size_Mat];
		  D.f[dirS] = &DD[dirN   *size_Mat];
		  D.f[dirN] = &DD[dirS   *size_Mat];
		  D.f[dirB] = &DD[dirT   *size_Mat];
		  D.f[dirT] = &DD[dirB   *size_Mat];
		  D.f[dirSW] = &DD[dirNE  *size_Mat];
		  D.f[dirNE] = &DD[dirSW  *size_Mat];
		  D.f[dirNW] = &DD[dirSE  *size_Mat];
		  D.f[dirSE] = &DD[dirNW  *size_Mat];
		  D.f[dirBW] = &DD[dirTE  *size_Mat];
		  D.f[dirTE] = &DD[dirBW  *size_Mat];
		  D.f[dirTW] = &DD[dirBE  *size_Mat];
		  D.f[dirBE] = &DD[dirTW  *size_Mat];
		  D.f[dirBS] = &DD[dirTN  *size_Mat];
		  D.f[dirTN] = &DD[dirBS  *size_Mat];
		  D.f[dirTS] = &DD[dirBN  *size_Mat];
		  D.f[dirBN] = &DD[dirTS  *size_Mat];
		  D.f[dirZERO] = &DD[dirZERO*size_Mat];
		  D.f[dirTNE] = &DD[dirBSW *size_Mat];
		  D.f[dirTSW] = &DD[dirBNE *size_Mat];
		  D.f[dirTSE] = &DD[dirBNW *size_Mat];
		  D.f[dirTNW] = &DD[dirBSE *size_Mat];
		  D.f[dirBNE] = &DD[dirTSW *size_Mat];
		  D.f[dirBSW] = &DD[dirTNE *size_Mat];
		  D.f[dirBSE] = &DD[dirTNW *size_Mat];
		  D.f[dirBNW] = &DD[dirTSE *size_Mat];
	  }
	  if (evenOrOdd==false)
      {
         DN.f[dirE   ] = &DD[dirE   *size_Mat];
         DN.f[dirW   ] = &DD[dirW   *size_Mat];
         DN.f[dirN   ] = &DD[dirN   *size_Mat];
         DN.f[dirS   ] = &DD[dirS   *size_Mat];
         DN.f[dirT   ] = &DD[dirT   *size_Mat];
         DN.f[dirB   ] = &DD[dirB   *size_Mat];
         DN.f[dirNE  ] = &DD[dirNE  *size_Mat];
         DN.f[dirSW  ] = &DD[dirSW  *size_Mat];
         DN.f[dirSE  ] = &DD[dirSE  *size_Mat];
         DN.f[dirNW  ] = &DD[dirNW  *size_Mat];
         DN.f[dirTE  ] = &DD[dirTE  *size_Mat];
         DN.f[dirBW  ] = &DD[dirBW  *size_Mat];
         DN.f[dirBE  ] = &DD[dirBE  *size_Mat];
         DN.f[dirTW  ] = &DD[dirTW  *size_Mat];
         DN.f[dirTN  ] = &DD[dirTN  *size_Mat];
         DN.f[dirBS  ] = &DD[dirBS  *size_Mat];
         DN.f[dirBN  ] = &DD[dirBN  *size_Mat];
         DN.f[dirTS  ] = &DD[dirTS  *size_Mat];
         DN.f[dirZERO] = &DD[dirZERO*size_Mat];
         DN.f[dirTNE ] = &DD[dirTNE *size_Mat];
         DN.f[dirTSW ] = &DD[dirTSW *size_Mat];
         DN.f[dirTSE ] = &DD[dirTSE *size_Mat];
         DN.f[dirTNW ] = &DD[dirTNW *size_Mat];
         DN.f[dirBNE ] = &DD[dirBNE *size_Mat];
         DN.f[dirBSW ] = &DD[dirBSW *size_Mat];
         DN.f[dirBSE ] = &DD[dirBSE *size_Mat];
         DN.f[dirBNW ] = &DD[dirBNW *size_Mat];
      } 
      else
      {
         DN.f[dirW   ] = &DD[dirE   *size_Mat];
         DN.f[dirE   ] = &DD[dirW   *size_Mat];
         DN.f[dirS   ] = &DD[dirN   *size_Mat];
         DN.f[dirN   ] = &DD[dirS   *size_Mat];
         DN.f[dirB   ] = &DD[dirT   *size_Mat];
         DN.f[dirT   ] = &DD[dirB   *size_Mat];
         DN.f[dirSW  ] = &DD[dirNE  *size_Mat];
         DN.f[dirNE  ] = &DD[dirSW  *size_Mat];
         DN.f[dirNW  ] = &DD[dirSE  *size_Mat];
         DN.f[dirSE  ] = &DD[dirNW  *size_Mat];
         DN.f[dirBW  ] = &DD[dirTE  *size_Mat];
         DN.f[dirTE  ] = &DD[dirBW  *size_Mat];
         DN.f[dirTW  ] = &DD[dirBE  *size_Mat];
         DN.f[dirBE  ] = &DD[dirTW  *size_Mat];
         DN.f[dirBS  ] = &DD[dirTN  *size_Mat];
         DN.f[dirTN  ] = &DD[dirBS  *size_Mat];
         DN.f[dirTS  ] = &DD[dirBN  *size_Mat];
         DN.f[dirBN  ] = &DD[dirTS  *size_Mat];
         DN.f[dirZERO] = &DD[dirZERO*size_Mat];
         DN.f[dirTNE ] = &DD[dirBSW *size_Mat];
         DN.f[dirTSW ] = &DD[dirBNE *size_Mat];
         DN.f[dirTSE ] = &DD[dirBNW *size_Mat];
         DN.f[dirTNW ] = &DD[dirBSE *size_Mat];
         DN.f[dirBNE ] = &DD[dirTSW *size_Mat];
         DN.f[dirBSW ] = &DD[dirTNE *size_Mat];
         DN.f[dirBSE ] = &DD[dirTNW *size_Mat];
         DN.f[dirBNW ] = &DD[dirTSE *size_Mat];
      }
      ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //directions allways exchange
	  //(-1 -1 -1) (-1  0  0) ( 0 -1  0) ( 0  0 -1) (-1 -1  0) (-1  0 -1) ( 0 -1 -1) ( 1  1 -1) ( 1 -1  1) (-1  1  1) ( 1 -1  0) ( 1  0 -1) ( 0  1 -1)
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  //directions exchange if solid neighbor
	  //( 1  1  1) ( 1  0  0) ( 0  1  0) ( 0  0  1) ( 1  1  0) ( 1  0  1) ( 0  1  1) (-1 -1  1) (-1  1 -1) ( 1 -1 -1) (-1  1  0) (-1  0  1) ( 0 -1  1)
	  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	  real q, tmp;
      q = q_dirE[k];   if (q>=zero && q<=one){ if (geom[kw  ] < GEO_FLUID){tmp = (DN.f[dirW  ])[kw  ]; (DN.f[dirW  ])[kw  ]=(D.f[dirW  ])[kw  ]; (D.f[dirW  ])[kw  ]=tmp;}}
	  q = q_dirW[k];   if (q>=zero && q<=one){                            {tmp = (DN.f[dirE  ])[ke  ]; (DN.f[dirE  ])[ke  ]=(D.f[dirE  ])[ke  ]; (D.f[dirE  ])[ke  ]=tmp;}}
      q = q_dirN[k];   if (q>=zero && q<=one){ if (geom[ks  ] < GEO_FLUID){tmp = (DN.f[dirS  ])[ks  ]; (DN.f[dirS  ])[ks  ]=(D.f[dirS  ])[ks  ]; (D.f[dirS  ])[ks  ]=tmp;}}
      q = q_dirS[k];   if (q>=zero && q<=one){                            {tmp = (DN.f[dirN  ])[kn  ]; (DN.f[dirN  ])[kn  ]=(D.f[dirN  ])[kn  ]; (D.f[dirN  ])[kn  ]=tmp;}}
      q = q_dirT[k];   if (q>=zero && q<=one){ if (geom[kb  ] < GEO_FLUID){tmp = (DN.f[dirB  ])[kb  ]; (DN.f[dirB  ])[kb  ]=(D.f[dirB  ])[kb  ]; (D.f[dirB  ])[kb  ]=tmp;}}
      q = q_dirB[k];   if (q>=zero && q<=one){                            {tmp = (DN.f[dirT  ])[kt  ]; (DN.f[dirT  ])[kt  ]=(D.f[dirT  ])[kt  ]; (D.f[dirT  ])[kt  ]=tmp;}}
      q = q_dirNE[k];  if (q>=zero && q<=one){ if (geom[ksw ] < GEO_FLUID){tmp = (DN.f[dirSW ])[ksw ]; (DN.f[dirSW ])[ksw ]=(D.f[dirSW ])[ksw ]; (D.f[dirSW ])[ksw ]=tmp;}}
      q = q_dirSW[k];  if (q>=zero && q<=one){                            {tmp = (DN.f[dirNE ])[kne ]; (DN.f[dirNE ])[kne ]=(D.f[dirNE ])[kne ]; (D.f[dirNE ])[kne ]=tmp;}}
      q = q_dirSE[k];  if (q>=zero && q<=one){                            {tmp = (DN.f[dirNW ])[knw ]; (DN.f[dirNW ])[knw ]=(D.f[dirNW ])[knw ]; (D.f[dirNW ])[knw ]=tmp;}}
      q = q_dirNW[k];  if (q>=zero && q<=one){ if (geom[kmp0] < GEO_FLUID){tmp = (DN.f[dirSE ])[kse ]; (DN.f[dirSE ])[kse ]=(D.f[dirSE ])[kse ]; (D.f[dirSE ])[kse ]=tmp;}}
      q = q_dirTE[k];  if (q>=zero && q<=one){ if (geom[kbw ] < GEO_FLUID){tmp = (DN.f[dirBW ])[kbw ]; (DN.f[dirBW ])[kbw ]=(D.f[dirBW ])[kbw ]; (D.f[dirBW ])[kbw ]=tmp;}}
      q = q_dirBW[k];  if (q>=zero && q<=one){                            {tmp = (DN.f[dirTE ])[kte ]; (DN.f[dirTE ])[kte ]=(D.f[dirTE ])[kte ]; (D.f[dirTE ])[kte ]=tmp;}}
      q = q_dirBE[k];  if (q>=zero && q<=one){                            {tmp = (DN.f[dirTW ])[ktw ]; (DN.f[dirTW ])[ktw ]=(D.f[dirTW ])[ktw ]; (D.f[dirTW ])[ktw ]=tmp;}}
      q = q_dirTW[k];  if (q>=zero && q<=one){ if (geom[km0p] < GEO_FLUID){tmp = (DN.f[dirBE ])[kbe ]; (DN.f[dirBE ])[kbe ]=(D.f[dirBE ])[kbe ]; (D.f[dirBE ])[kbe ]=tmp;}}
      q = q_dirTN[k];  if (q>=zero && q<=one){ if (geom[kbs ] < GEO_FLUID){tmp = (DN.f[dirBS ])[kbs ]; (DN.f[dirBS ])[kbs ]=(D.f[dirBS ])[kbs ]; (D.f[dirBS ])[kbs ]=tmp;}}
      q = q_dirBS[k];  if (q>=zero && q<=one){                            {tmp = (DN.f[dirTN ])[ktn ]; (DN.f[dirTN ])[ktn ]=(D.f[dirTN ])[ktn ]; (D.f[dirTN ])[ktn ]=tmp;}}
      q = q_dirBN[k];  if (q>=zero && q<=one){                            {tmp = (DN.f[dirTS ])[kts ]; (DN.f[dirTS ])[kts ]=(D.f[dirTS ])[kts ]; (D.f[dirTS ])[kts ]=tmp;}}
      q = q_dirTS[k];  if (q>=zero && q<=one){ if (geom[k0mp] < GEO_FLUID){tmp = (DN.f[dirBN ])[kbn ]; (DN.f[dirBN ])[kbn ]=(D.f[dirBN ])[kbn ]; (D.f[dirBN ])[kbn ]=tmp;}}
      q = q_dirTNE[k]; if (q>=zero && q<=one){ if (geom[kbsw] < GEO_FLUID){tmp = (DN.f[dirBSW])[kbsw]; (DN.f[dirBSW])[kbsw]=(D.f[dirBSW])[kbsw]; (D.f[dirBSW])[kbsw]=tmp;}}
      q = q_dirBSW[k]; if (q>=zero && q<=one){                            {tmp = (DN.f[dirTNE])[ktne]; (DN.f[dirTNE])[ktne]=(D.f[dirTNE])[ktne]; (D.f[dirTNE])[ktne]=tmp;}}
      q = q_dirBNE[k]; if (q>=zero && q<=one){                            {tmp = (DN.f[dirTSW])[ktsw]; (DN.f[dirTSW])[ktsw]=(D.f[dirTSW])[ktsw]; (D.f[dirTSW])[ktsw]=tmp;}}
      q = q_dirTSW[k]; if (q>=zero && q<=one){ if (geom[kmmp] < GEO_FLUID){tmp = (DN.f[dirBNE])[kbne]; (DN.f[dirBNE])[kbne]=(D.f[dirBNE])[kbne]; (D.f[dirBNE])[kbne]=tmp;}}
      q = q_dirTSE[k]; if (q>=zero && q<=one){                            {tmp = (DN.f[dirBNW])[kbnw]; (DN.f[dirBNW])[kbnw]=(D.f[dirBNW])[kbnw]; (D.f[dirBNW])[kbnw]=tmp;}}
      q = q_dirBNW[k]; if (q>=zero && q<=one){ if (geom[kmpm] < GEO_FLUID){tmp = (DN.f[dirTSE])[ktse]; (DN.f[dirTSE])[ktse]=(D.f[dirTSE])[ktse]; (D.f[dirTSE])[ktse]=tmp;}}
      q = q_dirBSE[k]; if (q>=zero && q<=one){ if (geom[kpmm] < GEO_FLUID){tmp = (DN.f[dirTNW])[ktnw]; (DN.f[dirTNW])[ktnw]=(D.f[dirTNW])[ktnw]; (D.f[dirTNW])[ktnw]=tmp;}}
      q = q_dirTNW[k]; if (q>=zero && q<=one){                            {tmp = (DN.f[dirBSE])[kbse]; (DN.f[dirBSE])[kbse]=(D.f[dirBSE])[kbse]; (D.f[dirBSE])[kbse]=tmp;}}
   }
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
