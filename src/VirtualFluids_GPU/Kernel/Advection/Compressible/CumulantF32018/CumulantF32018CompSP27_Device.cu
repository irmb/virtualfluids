#include "hip/hip_runtime.h"
#include "CumulantF32018CompSP27_Device.cuh"

#include "LBM/D3Q27.h"
#include "math.h"
#include "GPU/constant.h"

extern "C" __global__ void LB_Kernel_Cumulant_D3Q27F3_2018(real omega,
	unsigned int* bcMatD,
	unsigned int* neighborX,
	unsigned int* neighborY,
	unsigned int* neighborZ,
	real* DDStart,
	real* G6,
	int size_Mat,
	int level,
	real* forces,
	bool EvenOrOdd)
{
	////////////////////////////////////////////////////////////////////////////////
	const unsigned  x = threadIdx.x;  // Globaler x-Index 
	const unsigned  y = blockIdx.x;   // Globaler y-Index 
	const unsigned  z = blockIdx.y;   // Globaler z-Index 

	const unsigned nx = blockDim.x;
	const unsigned ny = gridDim.x;

	const unsigned k = nx*(ny*z + y) + x;
	//////////////////////////////////////////////////////////////////////////

	if (k < size_Mat)
	{
		////////////////////////////////////////////////////////////////////////////////
		unsigned int BC;
		BC = bcMatD[k];

		if ((BC != GEO_SOLID) && (BC != GEO_VOID))
		{
			Distributions27 D;
			if (EvenOrOdd == true)
			{
				D.f[dirE] = &DDStart[dirE   *size_Mat];
				D.f[dirW] = &DDStart[dirW   *size_Mat];
				D.f[dirN] = &DDStart[dirN   *size_Mat];
				D.f[dirS] = &DDStart[dirS   *size_Mat];
				D.f[dirT] = &DDStart[dirT   *size_Mat];
				D.f[dirB] = &DDStart[dirB   *size_Mat];
				D.f[dirNE] = &DDStart[dirNE  *size_Mat];
				D.f[dirSW] = &DDStart[dirSW  *size_Mat];
				D.f[dirSE] = &DDStart[dirSE  *size_Mat];
				D.f[dirNW] = &DDStart[dirNW  *size_Mat];
				D.f[dirTE] = &DDStart[dirTE  *size_Mat];
				D.f[dirBW] = &DDStart[dirBW  *size_Mat];
				D.f[dirBE] = &DDStart[dirBE  *size_Mat];
				D.f[dirTW] = &DDStart[dirTW  *size_Mat];
				D.f[dirTN] = &DDStart[dirTN  *size_Mat];
				D.f[dirBS] = &DDStart[dirBS  *size_Mat];
				D.f[dirBN] = &DDStart[dirBN  *size_Mat];
				D.f[dirTS] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirTNE] = &DDStart[dirTNE *size_Mat];
				D.f[dirTSW] = &DDStart[dirTSW *size_Mat];
				D.f[dirTSE] = &DDStart[dirTSE *size_Mat];
				D.f[dirTNW] = &DDStart[dirTNW *size_Mat];
				D.f[dirBNE] = &DDStart[dirBNE *size_Mat];
				D.f[dirBSW] = &DDStart[dirBSW *size_Mat];
				D.f[dirBSE] = &DDStart[dirBSE *size_Mat];
				D.f[dirBNW] = &DDStart[dirBNW *size_Mat];
			}
			else
			{
				D.f[dirW] = &DDStart[dirE   *size_Mat];
				D.f[dirE] = &DDStart[dirW   *size_Mat];
				D.f[dirS] = &DDStart[dirN   *size_Mat];
				D.f[dirN] = &DDStart[dirS   *size_Mat];
				D.f[dirB] = &DDStart[dirT   *size_Mat];
				D.f[dirT] = &DDStart[dirB   *size_Mat];
				D.f[dirSW] = &DDStart[dirNE  *size_Mat];
				D.f[dirNE] = &DDStart[dirSW  *size_Mat];
				D.f[dirNW] = &DDStart[dirSE  *size_Mat];
				D.f[dirSE] = &DDStart[dirNW  *size_Mat];
				D.f[dirBW] = &DDStart[dirTE  *size_Mat];
				D.f[dirTE] = &DDStart[dirBW  *size_Mat];
				D.f[dirTW] = &DDStart[dirBE  *size_Mat];
				D.f[dirBE] = &DDStart[dirTW  *size_Mat];
				D.f[dirBS] = &DDStart[dirTN  *size_Mat];
				D.f[dirTN] = &DDStart[dirBS  *size_Mat];
				D.f[dirTS] = &DDStart[dirBN  *size_Mat];
				D.f[dirBN] = &DDStart[dirTS  *size_Mat];
				D.f[dirZERO] = &DDStart[dirZERO*size_Mat];
				D.f[dirBSW] = &DDStart[dirTNE *size_Mat];
				D.f[dirBNE] = &DDStart[dirTSW *size_Mat];
				D.f[dirBNW] = &DDStart[dirTSE *size_Mat];
				D.f[dirBSE] = &DDStart[dirTNW *size_Mat];
				D.f[dirTSW] = &DDStart[dirBNE *size_Mat];
				D.f[dirTNE] = &DDStart[dirBSW *size_Mat];
				D.f[dirTNW] = &DDStart[dirBSE *size_Mat];
				D.f[dirTSE] = &DDStart[dirBNW *size_Mat];
			}

			Distributions6 G;
			if (EvenOrOdd == true)
			{
				G.g[dirE] = &G6[dirE   *size_Mat];
				G.g[dirW] = &G6[dirW   *size_Mat];
				G.g[dirN] = &G6[dirN   *size_Mat];
				G.g[dirS] = &G6[dirS   *size_Mat];
				G.g[dirT] = &G6[dirT   *size_Mat];
				G.g[dirB] = &G6[dirB   *size_Mat];
			}
			else
			{
				G.g[dirW] = &G6[dirE   *size_Mat];
				G.g[dirE] = &G6[dirW   *size_Mat];
				G.g[dirS] = &G6[dirN   *size_Mat];
				G.g[dirN] = &G6[dirS   *size_Mat];
				G.g[dirB] = &G6[dirT   *size_Mat];
				G.g[dirT] = &G6[dirB   *size_Mat];
			}

			////////////////////////////////////////////////////////////////////////////////
			//index
			//unsigned int kzero= k;
			//unsigned int ke   = k;
			unsigned int kw = neighborX[k];
			//unsigned int kn   = k;
			unsigned int ks = neighborY[k];
			//unsigned int kt   = k;
			unsigned int kb = neighborZ[k];
			unsigned int ksw = neighborY[kw];
			//unsigned int kne  = k;
			//unsigned int kse  = ks;
			//unsigned int knw  = kw;
			unsigned int kbw = neighborZ[kw];
			//unsigned int kte  = k;
			//unsigned int kbe  = kb;
			//unsigned int ktw  = kw;
			unsigned int kbs = neighborZ[ks];
			//unsigned int ktn  = k;
			//unsigned int kbn  = kb;
			//unsigned int kts  = ks;
			//unsigned int ktse = ks;
			//unsigned int kbnw = kbw;
			//unsigned int ktnw = kw;
			//unsigned int kbse = kbs;
			//unsigned int ktsw = ksw;
			//unsigned int kbne = kb;
			//unsigned int ktne = k;
			unsigned int kbsw = neighborZ[ksw];
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mgcbb = (G.g[dirE])[k];
			real mgabb = (G.g[dirW])[kw];
			real mgbcb = (G.g[dirN])[k];
			real mgbab = (G.g[dirS])[ks];
			real mgbbc = (G.g[dirT])[k];
			real mgbba = (G.g[dirB])[kb];
			real dxuxdxux = c1o2 * (-mgcbb + mgabb);
			real dyuydyuy = c1o2 * (-mgbcb + mgbab);
			real dzuzdzuz = c1o2 * (-mgbbc + mgbba);
			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			real mfcbb = (D.f[dirE])[k];
			real mfabb = (D.f[dirW])[kw];
			real mfbcb = (D.f[dirN])[k];
			real mfbab = (D.f[dirS])[ks];
			real mfbbc = (D.f[dirT])[k];
			real mfbba = (D.f[dirB])[kb];
			real mfccb = (D.f[dirNE])[k];
			real mfaab = (D.f[dirSW])[ksw];
			real mfcab = (D.f[dirSE])[ks];
			real mfacb = (D.f[dirNW])[kw];
			real mfcbc = (D.f[dirTE])[k];
			real mfaba = (D.f[dirBW])[kbw];
			real mfcba = (D.f[dirBE])[kb];
			real mfabc = (D.f[dirTW])[kw];
			real mfbcc = (D.f[dirTN])[k];
			real mfbaa = (D.f[dirBS])[kbs];
			real mfbca = (D.f[dirBN])[kb];
			real mfbac = (D.f[dirTS])[ks];
			real mfbbb = (D.f[dirZERO])[k];
			real mfccc = (D.f[dirTNE])[k];
			real mfaac = (D.f[dirTSW])[ksw];
			real mfcac = (D.f[dirTSE])[ks];
			real mfacc = (D.f[dirTNW])[kw];
			real mfcca = (D.f[dirBNE])[kb];
			real mfaaa = (D.f[dirBSW])[kbsw];
			real mfcaa = (D.f[dirBSE])[kbs];
			real mfaca = (D.f[dirBNW])[kbw];
			////////////////////////////////////////////////////////////////////////////////////
			real drho = ((((mfccc + mfaaa) + (mfaca + mfcac)) + ((mfacc + mfcaa) + (mfaac + mfcca))) +
				(((mfbac + mfbca) + (mfbaa + mfbcc)) + ((mfabc + mfcba) + (mfaba + mfcbc)) + ((mfacb + mfcab) + (mfaab + mfccb))) +
				((mfabb + mfcbb) + (mfbab + mfbcb) + (mfbba + mfbbc))) + mfbbb;

			real rho = one + drho;
			////////////////////////////////////////////////////////////////////////////////////
			real vvx = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfcaa - mfacc) + (mfcca - mfaac))) +
				(((mfcba - mfabc) + (mfcbc - mfaba)) + ((mfcab - mfacb) + (mfccb - mfaab))) +
				(mfcbb - mfabb)) / rho;
			real vvy = ((((mfccc - mfaaa) + (mfaca - mfcac)) + ((mfacc - mfcaa) + (mfcca - mfaac))) +
				(((mfbca - mfbac) + (mfbcc - mfbaa)) + ((mfacb - mfcab) + (mfccb - mfaab))) +
				(mfbcb - mfbab)) / rho;
			real vvz = ((((mfccc - mfaaa) + (mfcac - mfaca)) + ((mfacc - mfcaa) + (mfaac - mfcca))) +
				(((mfbac - mfbca) + (mfbcc - mfbaa)) + ((mfabc - mfcba) + (mfcbc - mfaba))) +
				(mfbbc - mfbba)) / rho;
			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			real fx = forces[0] / (pow(two, level)); //zero;//0.0032653/(pow(two,level)); //0.000000005;//(two/1600000.0) / 120.0; //
			real fy = forces[1] / (pow(two, level)); //zero;
			real fz = forces[2] / (pow(two, level)); //zero;
			vvx += fx;
			vvy += fy;
			vvz += fz;
			////////////////////////////////////////////////////////////////////////////////////
			real oMdrho = one; // comp special
			real m0, m1, m2;
			real vx2;
			real vy2;
			real vz2;
			vx2 = vvx*vvx;
			vy2 = vvy*vvy;
			vz2 = vvz*vvz;
			////////////////////////////////////////////////////////////////////////////////////
			real wadjust;
			real qudricLimitP = 0.01f;// * 0.0001f;
			real qudricLimitM = 0.01f;// * 0.0001f;
			real qudricLimitD = 0.01f;// * 0.001f;
										 ////////////////////////////////////////////////////////////////////////////////////
										 //Hin
										 ////////////////////////////////////////////////////////////////////////////////////
										 // mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36  Konditionieren
										 ////////////////////////////////////////////////////////////////////////////////////
										 // Z - Dir
			m2 = mfaaa + mfaac;
			m1 = mfaac - mfaaa;
			m0 = m2 + mfaab;
			mfaaa = m0;
			m0 += c1o36 * oMdrho;
			mfaab = m1 - m0 * vvz;
			mfaac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaba + mfabc;
			m1 = mfabc - mfaba;
			m0 = m2 + mfabb;
			mfaba = m0;
			m0 += c1o9 * oMdrho;
			mfabb = m1 - m0 * vvz;
			mfabc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaca + mfacc;
			m1 = mfacc - mfaca;
			m0 = m2 + mfacb;
			mfaca = m0;
			m0 += c1o36 * oMdrho;
			mfacb = m1 - m0 * vvz;
			mfacc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbaa + mfbac;
			m1 = mfbac - mfbaa;
			m0 = m2 + mfbab;
			mfbaa = m0;
			m0 += c1o9 * oMdrho;
			mfbab = m1 - m0 * vvz;
			mfbac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbba + mfbbc;
			m1 = mfbbc - mfbba;
			m0 = m2 + mfbbb;
			mfbba = m0;
			m0 += c4o9 * oMdrho;
			mfbbb = m1 - m0 * vvz;
			mfbbc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbca + mfbcc;
			m1 = mfbcc - mfbca;
			m0 = m2 + mfbcb;
			mfbca = m0;
			m0 += c1o9 * oMdrho;
			mfbcb = m1 - m0 * vvz;
			mfbcc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcaa + mfcac;
			m1 = mfcac - mfcaa;
			m0 = m2 + mfcab;
			mfcaa = m0;
			m0 += c1o36 * oMdrho;
			mfcab = m1 - m0 * vvz;
			mfcac = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcba + mfcbc;
			m1 = mfcbc - mfcba;
			m0 = m2 + mfcbb;
			mfcba = m0;
			m0 += c1o9 * oMdrho;
			mfcbb = m1 - m0 * vvz;
			mfcbc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcca + mfccc;
			m1 = mfccc - mfcca;
			m0 = m2 + mfccb;
			mfcca = m0;
			m0 += c1o36 * oMdrho;
			mfccb = m1 - m0 * vvz;
			mfccc = m2 - two*	m1 * vvz + vz2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit  1/6, 0, 1/18, 2/3, 0, 2/9, 1/6, 0, 1/18 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m2 = mfaaa + mfaca;
			m1 = mfaca - mfaaa;
			m0 = m2 + mfaba;
			mfaaa = m0;
			m0 += c1o6 * oMdrho;
			mfaba = m1 - m0 * vvy;
			mfaca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaab + mfacb;
			m1 = mfacb - mfaab;
			m0 = m2 + mfabb;
			mfaab = m0;
			mfabb = m1 - m0 * vvy;
			mfacb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaac + mfacc;
			m1 = mfacc - mfaac;
			m0 = m2 + mfabc;
			mfaac = m0;
			m0 += c1o18 * oMdrho;
			mfabc = m1 - m0 * vvy;
			mfacc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbaa + mfbca;
			m1 = mfbca - mfbaa;
			m0 = m2 + mfbba;
			mfbaa = m0;
			m0 += c2o3 * oMdrho;
			mfbba = m1 - m0 * vvy;
			mfbca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbab + mfbcb;
			m1 = mfbcb - mfbab;
			m0 = m2 + mfbbb;
			mfbab = m0;
			mfbbb = m1 - m0 * vvy;
			mfbcb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfbac + mfbcc;
			m1 = mfbcc - mfbac;
			m0 = m2 + mfbbc;
			mfbac = m0;
			m0 += c2o9 * oMdrho;
			mfbbc = m1 - m0 * vvy;
			mfbcc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcaa + mfcca;
			m1 = mfcca - mfcaa;
			m0 = m2 + mfcba;
			mfcaa = m0;
			m0 += c1o6 * oMdrho;
			mfcba = m1 - m0 * vvy;
			mfcca = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcab + mfccb;
			m1 = mfccb - mfcab;
			m0 = m2 + mfcbb;
			mfcab = m0;
			mfcbb = m1 - m0 * vvy;
			mfccb = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfcac + mfccc;
			m1 = mfccc - mfcac;
			m0 = m2 + mfcbc;
			mfcac = m0;
			m0 += c1o18 * oMdrho;
			mfcbc = m1 - m0 * vvy;
			mfccc = m2 - two*	m1 * vvy + vy2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			// mit     1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9		Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m2 = mfaaa + mfcaa;
			m1 = mfcaa - mfaaa;
			m0 = m2 + mfbaa;
			mfaaa = m0;
			m0 += one* oMdrho;
			mfbaa = m1 - m0 * vvx;
			mfcaa = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaba + mfcba;
			m1 = mfcba - mfaba;
			m0 = m2 + mfbba;
			mfaba = m0;
			mfbba = m1 - m0 * vvx;
			mfcba = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaca + mfcca;
			m1 = mfcca - mfaca;
			m0 = m2 + mfbca;
			mfaca = m0;
			m0 += c1o3 * oMdrho;
			mfbca = m1 - m0 * vvx;
			mfcca = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaab + mfcab;
			m1 = mfcab - mfaab;
			m0 = m2 + mfbab;
			mfaab = m0;
			mfbab = m1 - m0 * vvx;
			mfcab = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfabb + mfcbb;
			m1 = mfcbb - mfabb;
			m0 = m2 + mfbbb;
			mfabb = m0;
			mfbbb = m1 - m0 * vvx;
			mfcbb = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfacb + mfccb;
			m1 = mfccb - mfacb;
			m0 = m2 + mfbcb;
			mfacb = m0;
			mfbcb = m1 - m0 * vvx;
			mfccb = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfaac + mfcac;
			m1 = mfcac - mfaac;
			m0 = m2 + mfbac;
			mfaac = m0;
			m0 += c1o3 * oMdrho;
			mfbac = m1 - m0 * vvx;
			mfcac = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfabc + mfcbc;
			m1 = mfcbc - mfabc;
			m0 = m2 + mfbbc;
			mfabc = m0;
			mfbbc = m1 - m0 * vvx;
			mfcbc = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			m2 = mfacc + mfccc;
			m1 = mfccc - mfacc;
			m0 = m2 + mfbcc;
			mfacc = m0;
			m0 += c1o9 * oMdrho;
			mfbcc = m1 - m0 * vvx;
			mfccc = m2 - two*	m1 * vvx + vx2 * m0;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////

			////////////////////////////////////////////////////////////////////////////////////
			// Cumulants
			////////////////////////////////////////////////////////////////////////////////////
			real OxxPyyPzz = one;	//set the bulk viscosity one is high / two is very low and zero is (too) high ... (also called omega 2)

										////////////////////////////////////////////////////////////
										//3.
										//////////////////////////////
			real OxyyPxzz = eight*(-two + omega)*(one + two*omega) / (-eight - fourteen*omega + seven*omega*omega);//one;
			real OxyyMxzz = eight*(-two + omega)*(-seven + four*omega) / (fiftysix - fifty*omega + nine*omega*omega);//one;
			real Oxyz = twentyfour*(-two + omega)*(-two - seven*omega + three*omega*omega) / (fourtyeight + c152*omega - c130*omega*omega + twentynine*omega*omega*omega);//one;
																																											 ////////////////////////////////////////////////////////////
																																											 //4.
																																											 //////////////////////////////
			real O4 = one;
			//////////////////////////////
			//real O4        = omega;//TRT
			////////////////////////////////////////////////////////////
			//5.
			//////////////////////////////
			real O5 = one;
			////////////////////////////////////////////////////////////
			//6.
			//////////////////////////////
			real O6 = one;
			////////////////////////////////////////////////////////////


			//central moments to cumulants
			//4.
			real CUMcbb = mfcbb - ((mfcaa + c1o3) * mfabb + two * mfbba * mfbab) / rho;
			real CUMbcb = mfbcb - ((mfaca + c1o3) * mfbab + two * mfbba * mfabb) / rho;
			real CUMbbc = mfbbc - ((mfaac + c1o3) * mfbba + two * mfbab * mfabb) / rho;

			real CUMcca = mfcca - (((mfcaa * mfaca + two * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho - c1o9*(drho / rho));
			real CUMcac = mfcac - (((mfcaa * mfaac + two * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho - c1o9*(drho / rho));
			real CUMacc = mfacc - (((mfaac * mfaca + two * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho - c1o9*(drho / rho));

			//5.
			real CUMbcc = mfbcc - ((mfaac * mfbca + mfaca * mfbac + four * mfabb * mfbbb + two * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) / rho;
			real CUMcbc = mfcbc - ((mfaac * mfcba + mfcaa * mfabc + four * mfbab * mfbbb + two * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) / rho;
			real CUMccb = mfccb - ((mfcaa * mfacb + mfaca * mfcab + four * mfbba * mfbbb + two * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) / rho;

			//6.

			real CUMccc = mfccc + ((-four *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- four * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- two * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
				+ (four * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
					+ two * (mfcaa * mfaca * mfaac)
					+ sixteen *  mfbba * mfbab * mfabb) / (rho * rho)
				- c1o3 * (mfacc + mfcac + mfcca) / rho
				- c1o9 * (mfcaa + mfaca + mfaac) / rho
				+ (two * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
					+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3
				+ c1o27*((drho * drho - drho) / (rho*rho)));

			//2.
			// linear combinations
			real mxxPyyPzz = mfcaa + mfaca + mfaac;
			real mxxMyy = mfcaa - mfaca;
			real mxxMzz = mfcaa - mfaac;

			////////////////////////////////////////////////////////////////////////////
			real Dxy = -three*omega*mfbba;
			real Dxz = -three*omega*mfbab;
			real Dyz = -three*omega*mfabb;

			//3.
			// linear combinations

			real mxxyPyzz = mfcba + mfabc;
			real mxxyMyzz = mfcba - mfabc;

			real mxxzPyyz = mfcab + mfacb;
			real mxxzMyyz = mfcab - mfacb;

			real mxyyPxzz = mfbca + mfbac;
			real mxyyMxzz = mfbca - mfbac;

			///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			//incl. correction		(hat noch nicht so gut funktioniert...Optimierungsbedarf??)

			real dxux = c1o2 * (-omega) *(mxxMyy + mxxMzz) + c1o2 *  OxxPyyPzz * (mfaaa - mxxPyyPzz);
			real dyuy = dxux + omega * c3o2 * mxxMyy;
			real dzuz = dxux + omega * c3o2 * mxxMzz;

			mgabb = vvx*dxux;
			mgbab = vvy*dyuy;
			mgbba = vvz*dzuz;

			mgcbb = vvx*dxux;
			mgbcb = vvy*dyuy;
			mgbbc = vvz*dzuz;

			//relax
			mxxPyyPzz += OxxPyyPzz*(mfaaa - mxxPyyPzz) - three * (one - c1o2 * OxxPyyPzz) * (vx2 * dxux + vy2 * dyuy + vz2 * dzuz)
				+ (six - three * (omega + OxxPyyPzz) + omega * OxxPyyPzz) / (three * omega) *
				(dxuxdxux + dyuydyuy + dzuzdzuz);
			mxxMyy += omega * (-mxxMyy) - three * (one + c1o2 * (-omega)) * (vx2 * dxux - vy2 * dyuy)
				+ omega * (two*(one / omega - c1o2) * (one / omega - c1o2) - c1o6) * (dxuxdxux - dyuydyuy);
			mxxMzz += omega * (-mxxMzz) - three * (one + c1o2 * (-omega)) * (vx2 * dxux - vz2 * dzuz)
				+ omega * (two*(one / omega - c1o2) * (one / omega - c1o2) - c1o6) *(dxuxdxux - dzuzdzuz);

			///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			////no correction
			//mxxPyyPzz += OxxPyyPzz*(mfaaa-mxxPyyPzz);//-magicBulk*OxxPyyPzz;
			//mxxMyy    += -(-omega) * (-mxxMyy);
			//mxxMzz    += -(-omega) * (-mxxMzz);
			/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			mfabb += omega * (-mfabb);
			mfbab += omega * (-mfbab);
			mfbba += omega * (-mfbba);
			//////////////////////////////////////////////////////////////////////////

			// linear combinations back
			mfcaa = c1o3 * (mxxMyy + mxxMzz + mxxPyyPzz);
			mfaca = c1o3 * (-two*  mxxMyy + mxxMzz + mxxPyyPzz);
			mfaac = c1o3 * (mxxMyy - two* mxxMzz + mxxPyyPzz);


			//relax
			//////////////////////////////////////////////////////////////////////////
			//das ist der limiter
			wadjust = Oxyz + (one - Oxyz)*abs(mfbbb) / (abs(mfbbb) + qudricLimitD);
			mfbbb += wadjust * (-mfbbb);
			wadjust = OxyyPxzz + (one - OxyyPxzz)*abs(mxxyPyzz) / (abs(mxxyPyzz) + qudricLimitP);
			mxxyPyzz += wadjust * (-mxxyPyzz);
			wadjust = OxyyMxzz + (one - OxyyMxzz)*abs(mxxyMyzz) / (abs(mxxyMyzz) + qudricLimitM);
			mxxyMyzz += wadjust * (-mxxyMyzz);
			wadjust = OxyyPxzz + (one - OxyyPxzz)*abs(mxxzPyyz) / (abs(mxxzPyyz) + qudricLimitP);
			mxxzPyyz += wadjust * (-mxxzPyyz);
			wadjust = OxyyMxzz + (one - OxyyMxzz)*abs(mxxzMyyz) / (abs(mxxzMyyz) + qudricLimitM);
			mxxzMyyz += wadjust * (-mxxzMyyz);
			wadjust = OxyyPxzz + (one - OxyyPxzz)*abs(mxyyPxzz) / (abs(mxyyPxzz) + qudricLimitP);
			mxyyPxzz += wadjust * (-mxyyPxzz);
			wadjust = OxyyMxzz + (one - OxyyMxzz)*abs(mxyyMxzz) / (abs(mxyyMxzz) + qudricLimitM);
			mxyyMxzz += wadjust * (-mxyyMxzz);
			//////////////////////////////////////////////////////////////////////////
			//ohne limiter
			//mfbbb     += OxyyMxzz * (-mfbbb);
			//mxxyPyzz  += OxyyPxzz * (-mxxyPyzz);
			//mxxyMyzz  += OxyyMxzz * (-mxxyMyzz);
			//mxxzPyyz  += OxyyPxzz * (-mxxzPyyz);
			//mxxzMyyz  += OxyyMxzz * (-mxxzMyyz);
			//mxyyPxzz  += OxyyPxzz * (-mxyyPxzz);
			//mxyyMxzz  += OxyyMxzz * (-mxyyMxzz);
			//////////////////////////////////////////////////////////////////////////

			// linear combinations back
			mfcba = (mxxyMyzz + mxxyPyzz) * c1o2;
			mfabc = (-mxxyMyzz + mxxyPyzz) * c1o2;
			mfcab = (mxxzMyyz + mxxzPyyz) * c1o2;
			mfacb = (-mxxzMyyz + mxxzPyyz) * c1o2;
			mfbca = (mxyyMxzz + mxyyPxzz) * c1o2;
			mfbac = (-mxyyMxzz + mxyyPxzz) * c1o2;

			//4.
			//////////////////////////////////////////////////////////////////////////
			//mit limiter
			//	wadjust    = O4+(one-O4)*abs(CUMacc)/(abs(CUMacc)+qudricLimit);
			//CUMacc    += wadjust * (-CUMacc);
			//	wadjust    = O4+(one-O4)*abs(CUMcac)/(abs(CUMcac)+qudricLimit);
			//CUMcac    += wadjust * (-CUMcac); 
			//	wadjust    = O4+(one-O4)*abs(CUMcca)/(abs(CUMcca)+qudricLimit);
			//CUMcca    += wadjust * (-CUMcca); 

			//	wadjust    = O4+(one-O4)*abs(CUMbbc)/(abs(CUMbbc)+qudricLimit);
			//CUMbbc    += wadjust * (-CUMbbc); 
			//	wadjust    = O4+(one-O4)*abs(CUMbcb)/(abs(CUMbcb)+qudricLimit);
			//CUMbcb    += wadjust * (-CUMbcb); 
			//	wadjust    = O4+(one-O4)*abs(CUMcbb)/(abs(CUMcbb)+qudricLimit);
			//CUMcbb    += wadjust * (-CUMcbb); 
			//////////////////////////////////////////////////////////////////////////
			//ohne limiter
			//CUMacc += O4 * (-CUMacc);
			//CUMcac += O4 * (-CUMcac);
			//CUMcca += O4 * (-CUMcca);
			//CUMbbc += O4 * (-CUMbbc);
			//CUMbcb += O4 * (-CUMbcb);
			//CUMcbb += O4 * (-CUMcbb);
			CUMacc = -O4*(one / omega - c1o2)*(dyuy + dzuz)*c2o3 *(four + two*omega - three*omega*omega) / (two - seven*omega + five*omega*omega) + (one - O4) * (CUMacc);
			CUMcac = -O4*(one / omega - c1o2)*(dxux + dzuz)*c2o3 *(four + two*omega - three*omega*omega) / (two - seven*omega + five*omega*omega) + (one - O4) * (CUMcac);
			CUMcca = -O4*(one / omega - c1o2)*(dyuy + dxux)*c2o3 *(four + two*omega - three*omega*omega) / (two - seven*omega + five*omega*omega) + (one - O4) * (CUMcca);
			CUMbbc = -O4*(one / omega - c1o2)*Dxy*c1o3 *(four + twentyeight*omega - fourteen*omega*omega) / (six - twentyone*omega + fiveteen*omega*omega) + (one - O4) * (CUMbbc);
			CUMbcb = -O4*(one / omega - c1o2)*Dxz*c1o3 *(four + twentyeight*omega - fourteen*omega*omega) / (six - twentyone*omega + fiveteen*omega*omega) + (one - O4) * (CUMbcb);
			CUMcbb = -O4*(one / omega - c1o2)*Dyz*c1o3 *(four + twentyeight*omega - fourteen*omega*omega) / (six - twentyone*omega + fiveteen*omega*omega) + (one - O4) * (CUMcbb);
			//////////////////////////////////////////////////////////////////////////


			//5.
			CUMbcc += O5 * (-CUMbcc);
			CUMcbc += O5 * (-CUMcbc);
			CUMccb += O5 * (-CUMccb);

			//6.
			CUMccc += O6 * (-CUMccc);



			//back cumulants to central moments
			//4.
			mfcbb = CUMcbb + ((mfcaa + c1o3) * mfabb + two * mfbba * mfbab) / rho;
			mfbcb = CUMbcb + ((mfaca + c1o3) * mfbab + two * mfbba * mfabb) / rho;
			mfbbc = CUMbbc + ((mfaac + c1o3) * mfbba + two * mfbab * mfabb) / rho;

			mfcca = CUMcca + (((mfcaa * mfaca + two * mfbba * mfbba) + c1o3 * (mfcaa + mfaca)) / rho - c1o9*(drho / rho));
			mfcac = CUMcac + (((mfcaa * mfaac + two * mfbab * mfbab) + c1o3 * (mfcaa + mfaac)) / rho - c1o9*(drho / rho));
			mfacc = CUMacc + (((mfaac * mfaca + two * mfabb * mfabb) + c1o3 * (mfaac + mfaca)) / rho - c1o9*(drho / rho));

			//5.
			mfbcc = CUMbcc + ((mfaac * mfbca + mfaca * mfbac + four * mfabb * mfbbb + two * (mfbab * mfacb + mfbba * mfabc)) + c1o3 * (mfbca + mfbac)) / rho;
			mfcbc = CUMcbc + ((mfaac * mfcba + mfcaa * mfabc + four * mfbab * mfbbb + two * (mfabb * mfcab + mfbba * mfbac)) + c1o3 * (mfcba + mfabc)) / rho;
			mfccb = CUMccb + ((mfcaa * mfacb + mfaca * mfcab + four * mfbba * mfbbb + two * (mfbab * mfbca + mfabb * mfcba)) + c1o3 * (mfacb + mfcab)) / rho;

			//6.

			mfccc = CUMccc - ((-four *  mfbbb * mfbbb
				- (mfcaa * mfacc + mfaca * mfcac + mfaac * mfcca)
				- four * (mfabb * mfcbb + mfbab * mfbcb + mfbba * mfbbc)
				- two * (mfbca * mfbac + mfcba * mfabc + mfcab * mfacb)) / rho
				+ (four * (mfbab * mfbab * mfaca + mfabb * mfabb * mfcaa + mfbba * mfbba * mfaac)
					+ two * (mfcaa * mfaca * mfaac)
					+ sixteen *  mfbba * mfbab * mfabb) / (rho * rho)
				- c1o3 * (mfacc + mfcac + mfcca) / rho
				- c1o9 * (mfcaa + mfaca + mfaac) / rho
				+ (two * (mfbab * mfbab + mfabb * mfabb + mfbba * mfbba)
					+ (mfaac * mfaca + mfaac * mfcaa + mfaca * mfcaa) + c1o3 *(mfaac + mfaca + mfcaa)) / (rho * rho) * c2o3
				+ c1o27*((drho * drho - drho) / (rho*rho)));
			////////////////////////////////////////////////////////////////////////////////////

			////////////////////////////////////////////////////////////////////////////////////
			//the force be with you
			mfbaa = -mfbaa;
			mfaba = -mfaba;
			mfaab = -mfaab;
			////////////////////////////////////////////////////////////////////////////////////


			////////////////////////////////////////////////////////////////////////////////////
			//back
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1, 0, 1/3, 0, 0, 0, 1/3, 0, 1/9   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Z - Dir
			m0 = mfaac * c1o2 + mfaab * (vvz - c1o2) + (mfaaa + one* oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfaac - two* mfaab *  vvz + mfaaa                * (one - vz2) - one* oMdrho * vz2;
			m2 = mfaac * c1o2 + mfaab * (vvz + c1o2) + (mfaaa + one* oMdrho) * (vz2 + vvz) * c1o2;
			mfaaa = m0;
			mfaab = m1;
			mfaac = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfabc * c1o2 + mfabb * (vvz - c1o2) + mfaba * (vz2 - vvz) * c1o2;
			m1 = -mfabc - two* mfabb *  vvz + mfaba * (one - vz2);
			m2 = mfabc * c1o2 + mfabb * (vvz + c1o2) + mfaba * (vz2 + vvz) * c1o2;
			mfaba = m0;
			mfabb = m1;
			mfabc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacc * c1o2 + mfacb * (vvz - c1o2) + (mfaca + c1o3 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfacc - two* mfacb *  vvz + mfaca                  * (one - vz2) - c1o3 * oMdrho * vz2;
			m2 = mfacc * c1o2 + mfacb * (vvz + c1o2) + (mfaca + c1o3 * oMdrho) * (vz2 + vvz) * c1o2;
			mfaca = m0;
			mfacb = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfbac * c1o2 + mfbab * (vvz - c1o2) + mfbaa * (vz2 - vvz) * c1o2;
			m1 = -mfbac - two* mfbab *  vvz + mfbaa * (one - vz2);
			m2 = mfbac * c1o2 + mfbab * (vvz + c1o2) + mfbaa * (vz2 + vvz) * c1o2;
			mfbaa = m0;
			mfbab = m1;
			mfbac = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbbc * c1o2 + mfbbb * (vvz - c1o2) + mfbba * (vz2 - vvz) * c1o2;
			m1 = -mfbbc - two* mfbbb *  vvz + mfbba * (one - vz2);
			m2 = mfbbc * c1o2 + mfbbb * (vvz + c1o2) + mfbba * (vz2 + vvz) * c1o2;
			mfbba = m0;
			mfbbb = m1;
			mfbbc = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcc * c1o2 + mfbcb * (vvz - c1o2) + mfbca * (vz2 - vvz) * c1o2;
			m1 = -mfbcc - two* mfbcb *  vvz + mfbca * (one - vz2);
			m2 = mfbcc * c1o2 + mfbcb * (vvz + c1o2) + mfbca * (vz2 + vvz) * c1o2;
			mfbca = m0;
			mfbcb = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcac * c1o2 + mfcab * (vvz - c1o2) + (mfcaa + c1o3 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfcac - two* mfcab *  vvz + mfcaa                  * (one - vz2) - c1o3 * oMdrho * vz2;
			m2 = mfcac * c1o2 + mfcab * (vvz + c1o2) + (mfcaa + c1o3 * oMdrho) * (vz2 + vvz) * c1o2;
			mfcaa = m0;
			mfcab = m1;
			mfcac = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfcbc * c1o2 + mfcbb * (vvz - c1o2) + mfcba * (vz2 - vvz) * c1o2;
			m1 = -mfcbc - two* mfcbb *  vvz + mfcba * (one - vz2);
			m2 = mfcbc * c1o2 + mfcbb * (vvz + c1o2) + mfcba * (vz2 + vvz) * c1o2;
			mfcba = m0;
			mfcbb = m1;
			mfcbc = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfccb * (vvz - c1o2) + (mfcca + c1o9 * oMdrho) * (vz2 - vvz) * c1o2;
			m1 = -mfccc - two* mfccb *  vvz + mfcca                  * (one - vz2) - c1o9 * oMdrho * vz2;
			m2 = mfccc * c1o2 + mfccb * (vvz + c1o2) + (mfcca + c1o9 * oMdrho) * (vz2 + vvz) * c1o2;
			mfcca = m0;
			mfccb = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/6, 2/3, 1/6, 0, 0, 0, 1/18, 2/9, 1/18   Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// Y - Dir
			m0 = mfaca * c1o2 + mfaba * (vvy - c1o2) + (mfaaa + c1o6 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfaca - two* mfaba *  vvy + mfaaa                  * (one - vy2) - c1o6 * oMdrho * vy2;
			m2 = mfaca * c1o2 + mfaba * (vvy + c1o2) + (mfaaa + c1o6 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaaa = m0;
			mfaba = m1;
			mfaca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacb * c1o2 + mfabb * (vvy - c1o2) + (mfaab + c2o3 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfacb - two* mfabb *  vvy + mfaab                  * (one - vy2) - c2o3 * oMdrho * vy2;
			m2 = mfacb * c1o2 + mfabb * (vvy + c1o2) + (mfaab + c2o3 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaab = m0;
			mfabb = m1;
			mfacb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfacc * c1o2 + mfabc * (vvy - c1o2) + (mfaac + c1o6 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfacc - two* mfabc *  vvy + mfaac                  * (one - vy2) - c1o6 * oMdrho * vy2;
			m2 = mfacc * c1o2 + mfabc * (vvy + c1o2) + (mfaac + c1o6 * oMdrho) * (vy2 + vvy) * c1o2;
			mfaac = m0;
			mfabc = m1;
			mfacc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfbca * c1o2 + mfbba * (vvy - c1o2) + mfbaa * (vy2 - vvy) * c1o2;
			m1 = -mfbca - two* mfbba *  vvy + mfbaa * (one - vy2);
			m2 = mfbca * c1o2 + mfbba * (vvy + c1o2) + mfbaa * (vy2 + vvy) * c1o2;
			mfbaa = m0;
			mfbba = m1;
			mfbca = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcb * c1o2 + mfbbb * (vvy - c1o2) + mfbab * (vy2 - vvy) * c1o2;
			m1 = -mfbcb - two* mfbbb *  vvy + mfbab * (one - vy2);
			m2 = mfbcb * c1o2 + mfbbb * (vvy + c1o2) + mfbab * (vy2 + vvy) * c1o2;
			mfbab = m0;
			mfbbb = m1;
			mfbcb = m2;
			/////////b//////////////////////////////////////////////////////////////////////////
			m0 = mfbcc * c1o2 + mfbbc * (vvy - c1o2) + mfbac * (vy2 - vvy) * c1o2;
			m1 = -mfbcc - two* mfbbc *  vvy + mfbac * (one - vy2);
			m2 = mfbcc * c1o2 + mfbbc * (vvy + c1o2) + mfbac * (vy2 + vvy) * c1o2;
			mfbac = m0;
			mfbbc = m1;
			mfbcc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcca * c1o2 + mfcba * (vvy - c1o2) + (mfcaa + c1o18 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfcca - two* mfcba *  vvy + mfcaa                   * (one - vy2) - c1o18 * oMdrho * vy2;
			m2 = mfcca * c1o2 + mfcba * (vvy + c1o2) + (mfcaa + c1o18 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcaa = m0;
			mfcba = m1;
			mfcca = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccb * c1o2 + mfcbb * (vvy - c1o2) + (mfcab + c2o9 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfccb - two* mfcbb *  vvy + mfcab                  * (one - vy2) - c2o9 * oMdrho * vy2;
			m2 = mfccb * c1o2 + mfcbb * (vvy + c1o2) + (mfcab + c2o9 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcab = m0;
			mfcbb = m1;
			mfccb = m2;
			/////////c//////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfcbc * (vvy - c1o2) + (mfcac + c1o18 * oMdrho) * (vy2 - vvy) * c1o2;
			m1 = -mfccc - two* mfcbc *  vvy + mfcac                   * (one - vy2) - c1o18 * oMdrho * vy2;
			m2 = mfccc * c1o2 + mfcbc * (vvy + c1o2) + (mfcac + c1o18 * oMdrho) * (vy2 + vvy) * c1o2;
			mfcac = m0;
			mfcbc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			//mit 1/36, 1/9, 1/36, 1/9, 4/9, 1/9, 1/36, 1/9, 1/36 Konditionieren
			////////////////////////////////////////////////////////////////////////////////////
			// X - Dir
			m0 = mfcaa * c1o2 + mfbaa * (vvx - c1o2) + (mfaaa + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcaa - two* mfbaa *  vvx + mfaaa                   * (one - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcaa * c1o2 + mfbaa * (vvx + c1o2) + (mfaaa + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaaa = m0;
			mfbaa = m1;
			mfcaa = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcba * c1o2 + mfbba * (vvx - c1o2) + (mfaba + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcba - two* mfbba *  vvx + mfaba                  * (one - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcba * c1o2 + mfbba * (vvx + c1o2) + (mfaba + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaba = m0;
			mfbba = m1;
			mfcba = m2;
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcca * c1o2 + mfbca * (vvx - c1o2) + (mfaca + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcca - two* mfbca *  vvx + mfaca                   * (one - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcca * c1o2 + mfbca * (vvx + c1o2) + (mfaca + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaca = m0;
			mfbca = m1;
			mfcca = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcab * c1o2 + mfbab * (vvx - c1o2) + (mfaab + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcab - two* mfbab *  vvx + mfaab                  * (one - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcab * c1o2 + mfbab * (vvx + c1o2) + (mfaab + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaab = m0;
			mfbab = m1;
			mfcab = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 = mfcbb * c1o2 + mfbbb * (vvx - c1o2) + (mfabb + c4o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcbb - two* mfbbb *  vvx + mfabb                  * (one - vx2) - c4o9 * oMdrho * vx2;
			m2 = mfcbb * c1o2 + mfbbb * (vvx + c1o2) + (mfabb + c4o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfabb = m0;
			mfbbb = m1;
			mfcbb = m2;
			///////////b////////////////////////////////////////////////////////////////////////
			m0 = mfccb * c1o2 + mfbcb * (vvx - c1o2) + (mfacb + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfccb - two* mfbcb *  vvx + mfacb                  * (one - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfccb * c1o2 + mfbcb * (vvx + c1o2) + (mfacb + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfacb = m0;
			mfbcb = m1;
			mfccb = m2;
			////////////////////////////////////////////////////////////////////////////////////
			////////////////////////////////////////////////////////////////////////////////////
			m0 = mfcac * c1o2 + mfbac * (vvx - c1o2) + (mfaac + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcac - two* mfbac *  vvx + mfaac                   * (one - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfcac * c1o2 + mfbac * (vvx + c1o2) + (mfaac + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfaac = m0;
			mfbac = m1;
			mfcac = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 = mfcbc * c1o2 + mfbbc * (vvx - c1o2) + (mfabc + c1o9 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfcbc - two* mfbbc *  vvx + mfabc                  * (one - vx2) - c1o9 * oMdrho * vx2;
			m2 = mfcbc * c1o2 + mfbbc * (vvx + c1o2) + (mfabc + c1o9 * oMdrho) * (vx2 + vvx) * c1o2;
			mfabc = m0;
			mfbbc = m1;
			mfcbc = m2;
			///////////c////////////////////////////////////////////////////////////////////////
			m0 = mfccc * c1o2 + mfbcc * (vvx - c1o2) + (mfacc + c1o36 * oMdrho) * (vx2 - vvx) * c1o2;
			m1 = -mfccc - two* mfbcc *  vvx + mfacc                   * (one - vx2) - c1o36 * oMdrho * vx2;
			m2 = mfccc * c1o2 + mfbcc * (vvx + c1o2) + (mfacc + c1o36 * oMdrho) * (vx2 + vvx) * c1o2;
			mfacc = m0;
			mfbcc = m1;
			mfccc = m2;
			////////////////////////////////////////////////////////////////////////////////////

			////////////////////////////////////////////////////////////////////////////////////
			(D.f[dirE])[k] = mfabb;
			(D.f[dirW])[kw] = mfcbb;
			(D.f[dirN])[k] = mfbab;
			(D.f[dirS])[ks] = mfbcb;
			(D.f[dirT])[k] = mfbba;
			(D.f[dirB])[kb] = mfbbc;
			(D.f[dirNE])[k] = mfaab;
			(D.f[dirSW])[ksw] = mfccb;
			(D.f[dirSE])[ks] = mfacb;
			(D.f[dirNW])[kw] = mfcab;
			(D.f[dirTE])[k] = mfaba;
			(D.f[dirBW])[kbw] = mfcbc;
			(D.f[dirBE])[kb] = mfabc;
			(D.f[dirTW])[kw] = mfcba;
			(D.f[dirTN])[k] = mfbaa;
			(D.f[dirBS])[kbs] = mfbcc;
			(D.f[dirBN])[kb] = mfbac;
			(D.f[dirTS])[ks] = mfbca;
			(D.f[dirZERO])[k] = mfbbb;
			(D.f[dirTNE])[k] = mfaaa;
			(D.f[dirTSE])[ks] = mfaca;
			(D.f[dirBNE])[kb] = mfaac;
			(D.f[dirBSE])[kbs] = mfacc;
			(D.f[dirTNW])[kw] = mfcaa;
			(D.f[dirTSW])[ksw] = mfcca;
			(D.f[dirBNW])[kbw] = mfcac;
			(D.f[dirBSW])[kbsw] = mfccc;
			////////////////////////////////////////////////////////////////////////////////////

			(G.g[dirE])[k] = mgabb;
			(G.g[dirW])[kw] = mgcbb;
			(G.g[dirN])[k] = mgbab;
			(G.g[dirS])[ks] = mgbcb;
			(G.g[dirT])[k] = mgbba;
			(G.g[dirB])[kb] = mgbbc;
		}
	}
}